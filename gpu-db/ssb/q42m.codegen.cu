#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5e2bc5559940(uint64_t* COUNT5e2bc554a030, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5e2bc554a030, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5e2bc5559940(uint64_t* BUF_5e2bc554a030, uint64_t* BUF_IDX_5e2bc554a030, HASHTABLE_INSERT HT_5e2bc554a030, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e2bc554a030 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5e2bc554a030 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5e2bc554a030 = atomicAdd((int*)BUF_IDX_5e2bc554a030, 1);
HT_5e2bc554a030.insert(cuco::pair{KEY_5e2bc554a030, buf_idx_5e2bc554a030});
BUF_5e2bc554a030[buf_idx_5e2bc554a030 * 1 + 0] = tid;
}
__global__ void count_5e2bc54efbf0(uint64_t* COUNT5e2bc5549b00, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5e2bc5549b00, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5e2bc54efbf0(uint64_t* BUF_5e2bc5549b00, uint64_t* BUF_IDX_5e2bc5549b00, HASHTABLE_INSERT HT_5e2bc5549b00, DBI32Type* customer__c_custkey, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e2bc5549b00 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5e2bc5549b00 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_5e2bc5549b00 = atomicAdd((int*)BUF_IDX_5e2bc5549b00, 1);
HT_5e2bc5549b00.insert(cuco::pair{KEY_5e2bc5549b00, buf_idx_5e2bc5549b00});
BUF_5e2bc5549b00[buf_idx_5e2bc5549b00 * 1 + 0] = tid;
}
__global__ void count_5e2bc54efe30(uint64_t* COUNT5e2bc552ab60, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5e2bc552ab60, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5e2bc54efe30(uint64_t* BUF_5e2bc552ab60, uint64_t* BUF_IDX_5e2bc552ab60, HASHTABLE_INSERT HT_5e2bc552ab60, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e2bc552ab60 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5e2bc552ab60 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_5e2bc552ab60 = atomicAdd((int*)BUF_IDX_5e2bc552ab60, 1);
HT_5e2bc552ab60.insert(cuco::pair{KEY_5e2bc552ab60, buf_idx_5e2bc552ab60});
BUF_5e2bc552ab60[buf_idx_5e2bc552ab60 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5e2bc555e9c0(uint64_t* BUF_5e2bc552ab60, uint64_t* BUF_5e2bc5549b00, uint64_t* BUF_5e2bc554a030, uint64_t* COUNT5e2bc5547ee0, HASHTABLE_PROBE HT_5e2bc552ab60, HASHTABLE_PROBE HT_5e2bc5549b00, HASHTABLE_PROBE HT_5e2bc554a030, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e2bc554a030 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5e2bc554a030 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_5e2bc554a030 = HT_5e2bc554a030.find(KEY_5e2bc554a030);
if (SLOT_5e2bc554a030 == HT_5e2bc554a030.end()) return;
if (!(true)) return;
uint64_t KEY_5e2bc5549b00 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5e2bc5549b00 |= reg_lineorder__lo_custkey;
//Probe Hash table
auto SLOT_5e2bc5549b00 = HT_5e2bc5549b00.find(KEY_5e2bc5549b00);
if (SLOT_5e2bc5549b00 == HT_5e2bc5549b00.end()) return;
if (!(true)) return;
uint64_t KEY_5e2bc552ab60 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5e2bc552ab60 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_5e2bc552ab60 = HT_5e2bc552ab60.find(KEY_5e2bc552ab60);
if (SLOT_5e2bc552ab60 == HT_5e2bc552ab60.end()) return;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5e2bc5547ee0, 1);
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5e2bc555e9c0(uint64_t* BUF_5e2bc552ab60, uint64_t* BUF_5e2bc5547ee0, uint64_t* BUF_5e2bc5549b00, uint64_t* BUF_5e2bc554a030, uint64_t* BUF_IDX_5e2bc5547ee0, HASHTABLE_PROBE HT_5e2bc552ab60, HASHTABLE_INSERT HT_5e2bc5547ee0, HASHTABLE_PROBE HT_5e2bc5549b00, HASHTABLE_PROBE HT_5e2bc554a030, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e2bc554a030 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5e2bc554a030 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_5e2bc554a030 = HT_5e2bc554a030.find(KEY_5e2bc554a030);
if (SLOT_5e2bc554a030 == HT_5e2bc554a030.end()) return;
if (!(true)) return;
uint64_t KEY_5e2bc5549b00 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5e2bc5549b00 |= reg_lineorder__lo_custkey;
//Probe Hash table
auto SLOT_5e2bc5549b00 = HT_5e2bc5549b00.find(KEY_5e2bc5549b00);
if (SLOT_5e2bc5549b00 == HT_5e2bc5549b00.end()) return;
if (!(true)) return;
uint64_t KEY_5e2bc552ab60 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5e2bc552ab60 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_5e2bc552ab60 = HT_5e2bc552ab60.find(KEY_5e2bc552ab60);
if (SLOT_5e2bc552ab60 == HT_5e2bc552ab60.end()) return;
if (!(true)) return;
uint64_t KEY_5e2bc5547ee0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5e2bc5547ee0 |= reg_lineorder__lo_partkey;
// Insert hash table kernel;
auto buf_idx_5e2bc5547ee0 = atomicAdd((int*)BUF_IDX_5e2bc5547ee0, 1);
HT_5e2bc5547ee0.insert(cuco::pair{KEY_5e2bc5547ee0, buf_idx_5e2bc5547ee0});
BUF_5e2bc5547ee0[buf_idx_5e2bc5547ee0 * 4 + 0] = BUF_5e2bc552ab60[SLOT_5e2bc552ab60->second * 1 + 0];
BUF_5e2bc5547ee0[buf_idx_5e2bc5547ee0 * 4 + 1] = BUF_5e2bc5549b00[SLOT_5e2bc5549b00->second * 1 + 0];
BUF_5e2bc5547ee0[buf_idx_5e2bc5547ee0 * 4 + 2] = BUF_5e2bc554a030[SLOT_5e2bc554a030->second * 1 + 0];
BUF_5e2bc5547ee0[buf_idx_5e2bc5547ee0 * 4 + 3] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5e2bc555c160(uint64_t* BUF_5e2bc5547ee0, HASHTABLE_INSERT HT_5e2bc5500900, HASHTABLE_PROBE HT_5e2bc5547ee0, DBI32Type* date__d_year, DBI16Type* part__p_category_encoded, DBStringType* part__p_mfgr, DBI32Type* part__p_partkey, size_t part_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_mfgr = part__p_mfgr[tid];
if (!((evaluatePredicate(reg_part__p_mfgr, "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr, "MFGR#2", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e2bc5547ee0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_5e2bc5547ee0 |= reg_part__p_partkey;
//Probe Hash table
auto SLOT_5e2bc5547ee0 = HT_5e2bc5547ee0.find(KEY_5e2bc5547ee0);
if (SLOT_5e2bc5547ee0 == HT_5e2bc5547ee0.end()) return;
if (!(true)) return;
uint64_t KEY_5e2bc5500900 = 0;
auto reg_date__d_year = date__d_year[BUF_5e2bc5547ee0[SLOT_5e2bc5547ee0->second * 4 + 0]];

KEY_5e2bc5500900 |= reg_date__d_year;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_5e2bc5547ee0[SLOT_5e2bc5547ee0->second * 4 + 2]];
KEY_5e2bc5500900 <<= 16;
KEY_5e2bc5500900 |= reg_supplier__s_nation_encoded;
auto reg_part__p_category_encoded = part__p_category_encoded[tid];
KEY_5e2bc5500900 <<= 16;
KEY_5e2bc5500900 |= reg_part__p_category_encoded;
//Create aggregation hash table
HT_5e2bc5500900.insert(cuco::pair{KEY_5e2bc5500900, 1});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5e2bc555c160(uint64_t* BUF_5e2bc5547ee0, HASHTABLE_FIND HT_5e2bc5500900, HASHTABLE_PROBE HT_5e2bc5547ee0, DBI32Type* KEY_5e2bc5500900date__d_year, DBI16Type* KEY_5e2bc5500900part__p_category_encoded, DBI16Type* KEY_5e2bc5500900supplier__s_nation_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBDecimalType* lineorder__lo_revenue, DBDecimalType* lineorder__lo_supplycost, DBI16Type* part__p_category_encoded, DBStringType* part__p_mfgr, DBI32Type* part__p_partkey, size_t part_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_mfgr = part__p_mfgr[tid];
if (!((evaluatePredicate(reg_part__p_mfgr, "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr, "MFGR#2", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e2bc5547ee0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_5e2bc5547ee0 |= reg_part__p_partkey;
//Probe Hash table
auto SLOT_5e2bc5547ee0 = HT_5e2bc5547ee0.find(KEY_5e2bc5547ee0);
if (SLOT_5e2bc5547ee0 == HT_5e2bc5547ee0.end()) return;
if (!(true)) return;
uint64_t KEY_5e2bc5500900 = 0;
auto reg_date__d_year = date__d_year[BUF_5e2bc5547ee0[SLOT_5e2bc5547ee0->second * 4 + 0]];

KEY_5e2bc5500900 |= reg_date__d_year;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_5e2bc5547ee0[SLOT_5e2bc5547ee0->second * 4 + 2]];
KEY_5e2bc5500900 <<= 16;
KEY_5e2bc5500900 |= reg_supplier__s_nation_encoded;
auto reg_part__p_category_encoded = part__p_category_encoded[tid];
KEY_5e2bc5500900 <<= 16;
KEY_5e2bc5500900 |= reg_part__p_category_encoded;
//Aggregate in hashtable
auto buf_idx_5e2bc5500900 = HT_5e2bc5500900.find(KEY_5e2bc5500900)->second;
auto reg_lineorder__lo_supplycost = lineorder__lo_supplycost[BUF_5e2bc5547ee0[SLOT_5e2bc5547ee0->second * 4 + 3]];
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_5e2bc5547ee0[SLOT_5e2bc5547ee0->second * 4 + 3]];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_revenue) - (reg_lineorder__lo_supplycost);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5e2bc5500900], reg_map0__tmp_attr1);
KEY_5e2bc5500900date__d_year[buf_idx_5e2bc5500900] = reg_date__d_year;
KEY_5e2bc5500900supplier__s_nation_encoded[buf_idx_5e2bc5500900] = reg_supplier__s_nation_encoded;
KEY_5e2bc5500900part__p_category_encoded[buf_idx_5e2bc5500900] = reg_part__p_category_encoded;
}
__global__ void count_5e2bc556ea00(uint64_t* COUNT5e2bc54decf0, size_t COUNT5e2bc5500900) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5e2bc5500900) return;
//Materialize count
atomicAdd((int*)COUNT5e2bc54decf0, 1);
}
__global__ void main_5e2bc556ea00(size_t COUNT5e2bc5500900, DBDecimalType* MAT5e2bc54decf0aggr0__tmp_attr0, DBI32Type* MAT5e2bc54decf0date__d_year, DBI16Type* MAT5e2bc54decf0part__p_category_encoded, DBI16Type* MAT5e2bc54decf0supplier__s_nation_encoded, uint64_t* MAT_IDX5e2bc54decf0, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_category_encoded, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5e2bc5500900) return;
//Materialize buffers
auto mat_idx5e2bc54decf0 = atomicAdd((int*)MAT_IDX5e2bc54decf0, 1);
auto reg_date__d_year = date__d_year[tid];
MAT5e2bc54decf0date__d_year[mat_idx5e2bc54decf0] = reg_date__d_year;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[tid];
MAT5e2bc54decf0supplier__s_nation_encoded[mat_idx5e2bc54decf0] = reg_supplier__s_nation_encoded;
auto reg_part__p_category_encoded = part__p_category_encoded[tid];
MAT5e2bc54decf0part__p_category_encoded[mat_idx5e2bc54decf0] = reg_part__p_category_encoded;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5e2bc54decf0aggr0__tmp_attr0[mat_idx5e2bc54decf0] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT5e2bc554a030;
hipMalloc(&d_COUNT5e2bc554a030, sizeof(uint64_t));
hipMemset(d_COUNT5e2bc554a030, 0, sizeof(uint64_t));
count_5e2bc5559940<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT5e2bc554a030, d_supplier__s_region, supplier_size);
uint64_t COUNT5e2bc554a030;
hipMemcpy(&COUNT5e2bc554a030, d_COUNT5e2bc554a030, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5e2bc554a030;
hipMalloc(&d_BUF_IDX_5e2bc554a030, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5e2bc554a030, 0, sizeof(uint64_t));
uint64_t* d_BUF_5e2bc554a030;
hipMalloc(&d_BUF_5e2bc554a030, sizeof(uint64_t) * COUNT5e2bc554a030 * 1);
auto d_HT_5e2bc554a030 = cuco::static_map{ (int)COUNT5e2bc554a030*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5e2bc5559940<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_5e2bc554a030, d_BUF_IDX_5e2bc554a030, d_HT_5e2bc554a030.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5e2bc5549b00;
hipMalloc(&d_COUNT5e2bc5549b00, sizeof(uint64_t));
hipMemset(d_COUNT5e2bc5549b00, 0, sizeof(uint64_t));
count_5e2bc54efbf0<<<std::ceil((float)customer_size/128.), 128>>>(d_COUNT5e2bc5549b00, d_customer__c_region, customer_size);
uint64_t COUNT5e2bc5549b00;
hipMemcpy(&COUNT5e2bc5549b00, d_COUNT5e2bc5549b00, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5e2bc5549b00;
hipMalloc(&d_BUF_IDX_5e2bc5549b00, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5e2bc5549b00, 0, sizeof(uint64_t));
uint64_t* d_BUF_5e2bc5549b00;
hipMalloc(&d_BUF_5e2bc5549b00, sizeof(uint64_t) * COUNT5e2bc5549b00 * 1);
auto d_HT_5e2bc5549b00 = cuco::static_map{ (int)COUNT5e2bc5549b00*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5e2bc54efbf0<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_5e2bc5549b00, d_BUF_IDX_5e2bc5549b00, d_HT_5e2bc5549b00.ref(cuco::insert), d_customer__c_custkey, d_customer__c_region, customer_size);
//Materialize count
uint64_t* d_COUNT5e2bc552ab60;
hipMalloc(&d_COUNT5e2bc552ab60, sizeof(uint64_t));
hipMemset(d_COUNT5e2bc552ab60, 0, sizeof(uint64_t));
count_5e2bc54efe30<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT5e2bc552ab60, d_date__d_year, date_size);
uint64_t COUNT5e2bc552ab60;
hipMemcpy(&COUNT5e2bc552ab60, d_COUNT5e2bc552ab60, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5e2bc552ab60;
hipMalloc(&d_BUF_IDX_5e2bc552ab60, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5e2bc552ab60, 0, sizeof(uint64_t));
uint64_t* d_BUF_5e2bc552ab60;
hipMalloc(&d_BUF_5e2bc552ab60, sizeof(uint64_t) * COUNT5e2bc552ab60 * 1);
auto d_HT_5e2bc552ab60 = cuco::static_map{ (int)COUNT5e2bc552ab60*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5e2bc54efe30<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_5e2bc552ab60, d_BUF_IDX_5e2bc552ab60, d_HT_5e2bc552ab60.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Materialize count
uint64_t* d_COUNT5e2bc5547ee0;
hipMalloc(&d_COUNT5e2bc5547ee0, sizeof(uint64_t));
hipMemset(d_COUNT5e2bc5547ee0, 0, sizeof(uint64_t));
count_5e2bc555e9c0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5e2bc552ab60, d_BUF_5e2bc5549b00, d_BUF_5e2bc554a030, d_COUNT5e2bc5547ee0, d_HT_5e2bc552ab60.ref(cuco::find), d_HT_5e2bc5549b00.ref(cuco::find), d_HT_5e2bc554a030.ref(cuco::find), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size);
uint64_t COUNT5e2bc5547ee0;
hipMemcpy(&COUNT5e2bc5547ee0, d_COUNT5e2bc5547ee0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5e2bc5547ee0;
hipMalloc(&d_BUF_IDX_5e2bc5547ee0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5e2bc5547ee0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5e2bc5547ee0;
hipMalloc(&d_BUF_5e2bc5547ee0, sizeof(uint64_t) * COUNT5e2bc5547ee0 * 4);
auto d_HT_5e2bc5547ee0 = cuco::static_map{ (int)COUNT5e2bc5547ee0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5e2bc555e9c0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5e2bc552ab60, d_BUF_5e2bc5547ee0, d_BUF_5e2bc5549b00, d_BUF_5e2bc554a030, d_BUF_IDX_5e2bc5547ee0, d_HT_5e2bc552ab60.ref(cuco::find), d_HT_5e2bc5547ee0.ref(cuco::insert), d_HT_5e2bc5549b00.ref(cuco::find), d_HT_5e2bc554a030.ref(cuco::find), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
//Create aggregation hash table
auto d_HT_5e2bc5500900 = cuco::static_map{ (int)24650*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5e2bc555c160<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_5e2bc5547ee0, d_HT_5e2bc5500900.ref(cuco::insert), d_HT_5e2bc5547ee0.ref(cuco::find), d_date__d_year, d_part__p_category_encoded, d_part__p_mfgr, d_part__p_partkey, part_size, d_supplier__s_nation_encoded);
size_t COUNT5e2bc5500900 = d_HT_5e2bc5500900.size();
thrust::device_vector<int64_t> keys_5e2bc5500900(COUNT5e2bc5500900), vals_5e2bc5500900(COUNT5e2bc5500900);
d_HT_5e2bc5500900.retrieve_all(keys_5e2bc5500900.begin(), vals_5e2bc5500900.begin());
d_HT_5e2bc5500900.clear();
int64_t* raw_keys5e2bc5500900 = thrust::raw_pointer_cast(keys_5e2bc5500900.data());
insertKeys<<<std::ceil((float)COUNT5e2bc5500900/128.), 128>>>(raw_keys5e2bc5500900, d_HT_5e2bc5500900.ref(cuco::insert), COUNT5e2bc5500900);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5e2bc5500900);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5e2bc5500900);
DBI32Type* d_KEY_5e2bc5500900date__d_year;
hipMalloc(&d_KEY_5e2bc5500900date__d_year, sizeof(DBI32Type) * COUNT5e2bc5500900);
hipMemset(d_KEY_5e2bc5500900date__d_year, 0, sizeof(DBI32Type) * COUNT5e2bc5500900);
DBI16Type* d_KEY_5e2bc5500900supplier__s_nation_encoded;
hipMalloc(&d_KEY_5e2bc5500900supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT5e2bc5500900);
hipMemset(d_KEY_5e2bc5500900supplier__s_nation_encoded, 0, sizeof(DBI16Type) * COUNT5e2bc5500900);
DBI16Type* d_KEY_5e2bc5500900part__p_category_encoded;
hipMalloc(&d_KEY_5e2bc5500900part__p_category_encoded, sizeof(DBI16Type) * COUNT5e2bc5500900);
hipMemset(d_KEY_5e2bc5500900part__p_category_encoded, 0, sizeof(DBI16Type) * COUNT5e2bc5500900);
main_5e2bc555c160<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_5e2bc5547ee0, d_HT_5e2bc5500900.ref(cuco::find), d_HT_5e2bc5547ee0.ref(cuco::find), d_KEY_5e2bc5500900date__d_year, d_KEY_5e2bc5500900part__p_category_encoded, d_KEY_5e2bc5500900supplier__s_nation_encoded, d_aggr0__tmp_attr0, d_date__d_year, d_lineorder__lo_revenue, d_lineorder__lo_supplycost, d_part__p_category_encoded, d_part__p_mfgr, d_part__p_partkey, part_size, d_supplier__s_nation_encoded);
//Materialize count
uint64_t* d_COUNT5e2bc54decf0;
hipMalloc(&d_COUNT5e2bc54decf0, sizeof(uint64_t));
hipMemset(d_COUNT5e2bc54decf0, 0, sizeof(uint64_t));
count_5e2bc556ea00<<<std::ceil((float)COUNT5e2bc5500900/128.), 128>>>(d_COUNT5e2bc54decf0, COUNT5e2bc5500900);
uint64_t COUNT5e2bc54decf0;
hipMemcpy(&COUNT5e2bc54decf0, d_COUNT5e2bc54decf0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5e2bc54decf0;
hipMalloc(&d_MAT_IDX5e2bc54decf0, sizeof(uint64_t));
hipMemset(d_MAT_IDX5e2bc54decf0, 0, sizeof(uint64_t));
auto MAT5e2bc54decf0date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5e2bc54decf0);
DBI32Type* d_MAT5e2bc54decf0date__d_year;
hipMalloc(&d_MAT5e2bc54decf0date__d_year, sizeof(DBI32Type) * COUNT5e2bc54decf0);
auto MAT5e2bc54decf0supplier__s_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5e2bc54decf0);
DBI16Type* d_MAT5e2bc54decf0supplier__s_nation_encoded;
hipMalloc(&d_MAT5e2bc54decf0supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT5e2bc54decf0);
auto MAT5e2bc54decf0part__p_category_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5e2bc54decf0);
DBI16Type* d_MAT5e2bc54decf0part__p_category_encoded;
hipMalloc(&d_MAT5e2bc54decf0part__p_category_encoded, sizeof(DBI16Type) * COUNT5e2bc54decf0);
auto MAT5e2bc54decf0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5e2bc54decf0);
DBDecimalType* d_MAT5e2bc54decf0aggr0__tmp_attr0;
hipMalloc(&d_MAT5e2bc54decf0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5e2bc54decf0);
main_5e2bc556ea00<<<std::ceil((float)COUNT5e2bc5500900/128.), 128>>>(COUNT5e2bc5500900, d_MAT5e2bc54decf0aggr0__tmp_attr0, d_MAT5e2bc54decf0date__d_year, d_MAT5e2bc54decf0part__p_category_encoded, d_MAT5e2bc54decf0supplier__s_nation_encoded, d_MAT_IDX5e2bc54decf0, d_aggr0__tmp_attr0, d_KEY_5e2bc5500900date__d_year, d_KEY_5e2bc5500900part__p_category_encoded, d_KEY_5e2bc5500900supplier__s_nation_encoded);
hipMemcpy(MAT5e2bc54decf0date__d_year, d_MAT5e2bc54decf0date__d_year, sizeof(DBI32Type) * COUNT5e2bc54decf0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5e2bc54decf0supplier__s_nation_encoded, d_MAT5e2bc54decf0supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT5e2bc54decf0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5e2bc54decf0part__p_category_encoded, d_MAT5e2bc54decf0part__p_category_encoded, sizeof(DBI16Type) * COUNT5e2bc54decf0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5e2bc54decf0aggr0__tmp_attr0, d_MAT5e2bc54decf0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5e2bc54decf0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5e2bc54decf0; i++) { std::cout << "" << MAT5e2bc54decf0date__d_year[i];
std::cout << "," << supplier__s_nation_map[MAT5e2bc54decf0supplier__s_nation_encoded[i]];
std::cout << "," << part__p_category_map[MAT5e2bc54decf0part__p_category_encoded[i]];
std::cout << "," << MAT5e2bc54decf0aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_5e2bc554a030);
hipFree(d_BUF_IDX_5e2bc554a030);
hipFree(d_COUNT5e2bc554a030);
hipFree(d_BUF_5e2bc5549b00);
hipFree(d_BUF_IDX_5e2bc5549b00);
hipFree(d_COUNT5e2bc5549b00);
hipFree(d_BUF_5e2bc552ab60);
hipFree(d_BUF_IDX_5e2bc552ab60);
hipFree(d_COUNT5e2bc552ab60);
hipFree(d_BUF_5e2bc5547ee0);
hipFree(d_BUF_IDX_5e2bc5547ee0);
hipFree(d_COUNT5e2bc5547ee0);
hipFree(d_KEY_5e2bc5500900date__d_year);
hipFree(d_KEY_5e2bc5500900part__p_category_encoded);
hipFree(d_KEY_5e2bc5500900supplier__s_nation_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5e2bc54decf0);
hipFree(d_MAT5e2bc54decf0aggr0__tmp_attr0);
hipFree(d_MAT5e2bc54decf0date__d_year);
hipFree(d_MAT5e2bc54decf0part__p_category_encoded);
hipFree(d_MAT5e2bc54decf0supplier__s_nation_encoded);
hipFree(d_MAT_IDX5e2bc54decf0);
free(MAT5e2bc54decf0aggr0__tmp_attr0);
free(MAT5e2bc54decf0date__d_year);
free(MAT5e2bc54decf0part__p_category_encoded);
free(MAT5e2bc54decf0supplier__s_nation_encoded);
}