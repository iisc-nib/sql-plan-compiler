#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
__global__ void count_1(uint64_t* COUNT0, DBStringType* supplier__s_nation, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT0, 1);
}
template<typename HASHTABLE_INSERT_PK>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT_PK HT_0, DBStringType* supplier__s_nation, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0, buf_idx_0});
BUF_0[buf_idx_0 * 1 + 0] = tid;
}
__global__ void count_3(uint64_t* COUNT2, DBStringType* part__p_category, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_category = part__p_category[tid];
if (!(evaluatePredicate(reg_part__p_category, "MFGR#14", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT2, 1);
}
template<typename HASHTABLE_INSERT_PK>
__global__ void main_3(uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_INSERT_PK HT_2, DBStringType* part__p_category, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_category = part__p_category[tid];
if (!(evaluatePredicate(reg_part__p_category, "MFGR#14", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_2 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_2 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_2 = atomicAdd((int*)BUF_IDX_2, 1);
HT_2.insert(cuco::pair{KEY_2, buf_idx_2});
BUF_2[buf_idx_2 * 1 + 0] = tid;
}
__global__ void count_5(uint64_t* COUNT4, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT4, 1);
}
template<typename HASHTABLE_INSERT_PK>
__global__ void main_5(uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_INSERT_PK HT_4, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_4 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_4 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_4 = atomicAdd((int*)BUF_IDX_4, 1);
HT_4.insert(cuco::pair{KEY_4, buf_idx_4});
BUF_4[buf_idx_4 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE_PK>
__global__ void count_7(uint64_t* BUF_0, uint64_t* BUF_2, uint64_t* BUF_4, uint64_t* COUNT6, HASHTABLE_PROBE_PK HT_0, HASHTABLE_PROBE_PK HT_2, HASHTABLE_PROBE_PK HT_4, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_0 = HT_0.find(KEY_0);
if (SLOT_0 == HT_0.end()) return;
if (!(true)) return;
uint64_t KEY_2 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_2 |= reg_lineorder__lo_partkey;
//Probe Hash table
auto SLOT_2 = HT_2.find(KEY_2);
if (SLOT_2 == HT_2.end()) return;
if (!(true)) return;
uint64_t KEY_4 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_4 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_4 = HT_4.find(KEY_4);
if (SLOT_4 == HT_4.end()) return;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT6, 1);
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_INSERT>
__global__ void main_7(uint64_t* BUF_0, uint64_t* BUF_2, uint64_t* BUF_4, uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_PROBE_PK HT_0, HASHTABLE_PROBE_PK HT_2, HASHTABLE_PROBE_PK HT_4, HASHTABLE_INSERT HT_6, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_0 = HT_0.find(KEY_0);
if (SLOT_0 == HT_0.end()) return;
if (!(true)) return;
uint64_t KEY_2 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_2 |= reg_lineorder__lo_partkey;
//Probe Hash table
auto SLOT_2 = HT_2.find(KEY_2);
if (SLOT_2 == HT_2.end()) return;
if (!(true)) return;
uint64_t KEY_4 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_4 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_4 = HT_4.find(KEY_4);
if (SLOT_4 == HT_4.end()) return;
if (!(true)) return;
uint64_t KEY_6 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_6 |= reg_lineorder__lo_custkey;
// Insert hash table kernel;
auto buf_idx_6 = atomicAdd((int*)BUF_IDX_6, 1);
HT_6.insert(cuco::pair{KEY_6, buf_idx_6});
BUF_6[buf_idx_6 * 4 + 0] = BUF_4[SLOT_4->second * 1 + 0];
BUF_6[buf_idx_6 * 4 + 1] = BUF_0[SLOT_0->second * 1 + 0];
BUF_6[buf_idx_6 * 4 + 2] = BUF_2[SLOT_2->second * 1 + 0];
BUF_6[buf_idx_6 * 4 + 3] = tid;
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_9(uint64_t* BUF_6, HASHTABLE_PROBE HT_6, HASHTABLE_INSERT HT_8, DBI32Type* customer__c_custkey, size_t customer_size, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_6 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_6 |= reg_customer__c_custkey;
//Probe Hash table
HT_6.for_each(KEY_6, [&] __device__ (auto const SLOT_6) {

auto const [slot_first6, slot_second6] = SLOT_6;
if (!(true)) return;
uint64_t KEY_8 = 0;
auto reg_date__d_year = date__d_year[BUF_6[slot_second6 * 4 + 0]];

KEY_8 |= reg_date__d_year;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_6[slot_second6 * 4 + 1]];
KEY_8 <<= 16;
KEY_8 |= reg_supplier__s_city_encoded;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_6[slot_second6 * 4 + 2]];
KEY_8 <<= 16;
KEY_8 |= reg_part__p_brand1_encoded;
//Create aggregation hash table
HT_8.insert(cuco::pair{KEY_8, 1});
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_9(uint64_t* BUF_6, HASHTABLE_PROBE HT_6, HASHTABLE_FIND HT_8, DBI32Type* KEY_8date__d_year, DBI16Type* KEY_8part__p_brand1_encoded, DBI16Type* KEY_8supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* customer__c_custkey, size_t customer_size, DBI32Type* date__d_year, DBDecimalType* lineorder__lo_revenue, DBDecimalType* lineorder__lo_supplycost, DBI16Type* part__p_brand1_encoded, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_6 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_6 |= reg_customer__c_custkey;
//Probe Hash table
HT_6.for_each(KEY_6, [&] __device__ (auto const SLOT_6) {
auto const [slot_first6, slot_second6] = SLOT_6;
if (!(true)) return;
uint64_t KEY_8 = 0;
auto reg_date__d_year = date__d_year[BUF_6[slot_second6 * 4 + 0]];

KEY_8 |= reg_date__d_year;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_6[slot_second6 * 4 + 1]];
KEY_8 <<= 16;
KEY_8 |= reg_supplier__s_city_encoded;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_6[slot_second6 * 4 + 2]];
KEY_8 <<= 16;
KEY_8 |= reg_part__p_brand1_encoded;
//Aggregate in hashtable
auto buf_idx_8 = HT_8.find(KEY_8)->second;
auto reg_lineorder__lo_supplycost = lineorder__lo_supplycost[BUF_6[slot_second6 * 4 + 3]];
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_6[slot_second6 * 4 + 3]];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_revenue) - (reg_lineorder__lo_supplycost);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_8], reg_map0__tmp_attr1);
KEY_8date__d_year[buf_idx_8] = reg_date__d_year;
KEY_8supplier__s_city_encoded[buf_idx_8] = reg_supplier__s_city_encoded;
KEY_8part__p_brand1_encoded[buf_idx_8] = reg_part__p_brand1_encoded;
});
}
__global__ void count_11(uint64_t* COUNT10, size_t COUNT8) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT8) return;
//Materialize count
atomicAdd((int*)COUNT10, 1);
}
__global__ void main_11(size_t COUNT8, DBDecimalType* MAT10aggr0__tmp_attr0, DBI32Type* MAT10date__d_year, DBI16Type* MAT10part__p_brand1_encoded, DBI16Type* MAT10supplier__s_city_encoded, uint64_t* MAT_IDX10, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT8) return;
//Materialize buffers
auto mat_idx10 = atomicAdd((int*)MAT_IDX10, 1);
auto reg_date__d_year = date__d_year[tid];
MAT10date__d_year[mat_idx10] = reg_date__d_year;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[tid];
MAT10supplier__s_city_encoded[mat_idx10] = reg_supplier__s_city_encoded;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[tid];
MAT10part__p_brand1_encoded[mat_idx10] = reg_part__p_brand1_encoded;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT10aggr0__tmp_attr0[mat_idx10] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
size_t used_mem = usedGpuMem();
auto start = std::chrono::high_resolution_clock::now();
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT0, d_supplier__s_nation, supplier_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_supplier__s_nation, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT2;
hipMalloc(&d_COUNT2, sizeof(uint64_t));
hipMemset(d_COUNT2, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)part_size/128.), 128>>>(d_COUNT2, d_part__p_category, part_size);
uint64_t COUNT2;
hipMemcpy(&COUNT2, d_COUNT2, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 1);
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_2, d_BUF_IDX_2, d_HT_2.ref(cuco::insert), d_part__p_category, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT4;
hipMalloc(&d_COUNT4, sizeof(uint64_t));
hipMemset(d_COUNT4, 0, sizeof(uint64_t));
count_5<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT4, d_date__d_year, date_size);
uint64_t COUNT4;
hipMemcpy(&COUNT4, d_COUNT4, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 1);
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_4, d_BUF_IDX_4, d_HT_4.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Materialize count
uint64_t* d_COUNT6;
hipMalloc(&d_COUNT6, sizeof(uint64_t));
hipMemset(d_COUNT6, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_0, d_BUF_2, d_BUF_4, d_COUNT6, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::find), d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
uint64_t COUNT6;
hipMemcpy(&COUNT6, d_COUNT6, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 4);
auto d_HT_6 = cuco::experimental::static_multimap{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_0, d_BUF_2, d_BUF_4, d_BUF_6, d_BUF_IDX_6, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::insert), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
//Create aggregation hash table
auto d_HT_8 = cuco::static_map{ (int)2259*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_9<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_6, d_HT_6.ref(cuco::for_each), d_HT_8.ref(cuco::insert), d_customer__c_custkey, customer_size, d_date__d_year, d_part__p_brand1_encoded, d_supplier__s_city_encoded);
size_t COUNT8 = d_HT_8.size();
thrust::device_vector<int64_t> keys_8(COUNT8), vals_8(COUNT8);
d_HT_8.retrieve_all(keys_8.begin(), vals_8.begin());
d_HT_8.clear();
int64_t* raw_keys8 = thrust::raw_pointer_cast(keys_8.data());
insertKeys<<<std::ceil((float)COUNT8/128.), 128>>>(raw_keys8, d_HT_8.ref(cuco::insert), COUNT8);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT8);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT8);
DBI32Type* d_KEY_8date__d_year;
hipMalloc(&d_KEY_8date__d_year, sizeof(DBI32Type) * COUNT8);
hipMemset(d_KEY_8date__d_year, 0, sizeof(DBI32Type) * COUNT8);
DBI16Type* d_KEY_8supplier__s_city_encoded;
hipMalloc(&d_KEY_8supplier__s_city_encoded, sizeof(DBI16Type) * COUNT8);
hipMemset(d_KEY_8supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT8);
DBI16Type* d_KEY_8part__p_brand1_encoded;
hipMalloc(&d_KEY_8part__p_brand1_encoded, sizeof(DBI16Type) * COUNT8);
hipMemset(d_KEY_8part__p_brand1_encoded, 0, sizeof(DBI16Type) * COUNT8);
main_9<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_6, d_HT_6.ref(cuco::for_each), d_HT_8.ref(cuco::find), d_KEY_8date__d_year, d_KEY_8part__p_brand1_encoded, d_KEY_8supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_custkey, customer_size, d_date__d_year, d_lineorder__lo_revenue, d_lineorder__lo_supplycost, d_part__p_brand1_encoded, d_supplier__s_city_encoded);
//Materialize count
uint64_t* d_COUNT10;
hipMalloc(&d_COUNT10, sizeof(uint64_t));
hipMemset(d_COUNT10, 0, sizeof(uint64_t));
count_11<<<std::ceil((float)COUNT8/128.), 128>>>(d_COUNT10, COUNT8);
uint64_t COUNT10;
hipMemcpy(&COUNT10, d_COUNT10, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX10;
hipMalloc(&d_MAT_IDX10, sizeof(uint64_t));
hipMemset(d_MAT_IDX10, 0, sizeof(uint64_t));
auto MAT10date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT10);
DBI32Type* d_MAT10date__d_year;
hipMalloc(&d_MAT10date__d_year, sizeof(DBI32Type) * COUNT10);
auto MAT10supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT10);
DBI16Type* d_MAT10supplier__s_city_encoded;
hipMalloc(&d_MAT10supplier__s_city_encoded, sizeof(DBI16Type) * COUNT10);
auto MAT10part__p_brand1_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT10);
DBI16Type* d_MAT10part__p_brand1_encoded;
hipMalloc(&d_MAT10part__p_brand1_encoded, sizeof(DBI16Type) * COUNT10);
auto MAT10aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT10);
DBDecimalType* d_MAT10aggr0__tmp_attr0;
hipMalloc(&d_MAT10aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT10);
main_11<<<std::ceil((float)COUNT8/128.), 128>>>(COUNT8, d_MAT10aggr0__tmp_attr0, d_MAT10date__d_year, d_MAT10part__p_brand1_encoded, d_MAT10supplier__s_city_encoded, d_MAT_IDX10, d_aggr0__tmp_attr0, d_KEY_8date__d_year, d_KEY_8part__p_brand1_encoded, d_KEY_8supplier__s_city_encoded);
hipMemcpy(MAT10date__d_year, d_MAT10date__d_year, sizeof(DBI32Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10supplier__s_city_encoded, d_MAT10supplier__s_city_encoded, sizeof(DBI16Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10part__p_brand1_encoded, d_MAT10part__p_brand1_encoded, sizeof(DBI16Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10aggr0__tmp_attr0, d_MAT10aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT10, hipMemcpyDeviceToHost);
auto end = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < COUNT10; i++) { std::cout << "" << MAT10date__d_year[i];
std::cout << "|" << supplier__s_city_map[MAT10supplier__s_city_encoded[i]];
std::cout << "|" << part__p_brand1_map[MAT10part__p_brand1_encoded[i]];
std::cout << "|" << MAT10aggr0__tmp_attr0[i];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_COUNT0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_COUNT2);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_COUNT4);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_COUNT6);
hipFree(d_KEY_8date__d_year);
hipFree(d_KEY_8part__p_brand1_encoded);
hipFree(d_KEY_8supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT10);
hipFree(d_MAT10aggr0__tmp_attr0);
hipFree(d_MAT10date__d_year);
hipFree(d_MAT10part__p_brand1_encoded);
hipFree(d_MAT10supplier__s_city_encoded);
hipFree(d_MAT_IDX10);
free(MAT10aggr0__tmp_attr0);
free(MAT10date__d_year);
free(MAT10part__p_brand1_encoded);
free(MAT10supplier__s_city_encoded);
}