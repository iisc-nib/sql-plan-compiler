#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5e99fbeeaad0(uint64_t* COUNT5e99fbf47bf0, DBStringType* supplier__s_nation, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5e99fbf47bf0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5e99fbeeaad0(uint64_t* BUF_5e99fbf47bf0, uint64_t* BUF_IDX_5e99fbf47bf0, HASHTABLE_INSERT HT_5e99fbf47bf0, DBStringType* supplier__s_nation, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e99fbf47bf0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5e99fbf47bf0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5e99fbf47bf0 = atomicAdd((int*)BUF_IDX_5e99fbf47bf0, 1);
HT_5e99fbf47bf0.insert(cuco::pair{KEY_5e99fbf47bf0, buf_idx_5e99fbf47bf0});
BUF_5e99fbf47bf0[buf_idx_5e99fbf47bf0 * 1 + 0] = tid;
}
__global__ void count_5e99fbf55950(uint64_t* COUNT5e99fbf23fe0, DBStringType* part__p_category, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_category = part__p_category[tid];
if (!(evaluatePredicate(reg_part__p_category, "MFGR#14", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5e99fbf23fe0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5e99fbf55950(uint64_t* BUF_5e99fbf23fe0, uint64_t* BUF_IDX_5e99fbf23fe0, HASHTABLE_INSERT HT_5e99fbf23fe0, DBStringType* part__p_category, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_category = part__p_category[tid];
if (!(evaluatePredicate(reg_part__p_category, "MFGR#14", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e99fbf23fe0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_5e99fbf23fe0 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_5e99fbf23fe0 = atomicAdd((int*)BUF_IDX_5e99fbf23fe0, 1);
HT_5e99fbf23fe0.insert(cuco::pair{KEY_5e99fbf23fe0, buf_idx_5e99fbf23fe0});
BUF_5e99fbf23fe0[buf_idx_5e99fbf23fe0 * 1 + 0] = tid;
}
__global__ void count_5e99fbf450d0(uint64_t* COUNT5e99fbf41e40, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5e99fbf41e40, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5e99fbf450d0(uint64_t* BUF_5e99fbf41e40, uint64_t* BUF_IDX_5e99fbf41e40, HASHTABLE_INSERT HT_5e99fbf41e40, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e99fbf41e40 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5e99fbf41e40 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_5e99fbf41e40 = atomicAdd((int*)BUF_IDX_5e99fbf41e40, 1);
HT_5e99fbf41e40.insert(cuco::pair{KEY_5e99fbf41e40, buf_idx_5e99fbf41e40});
BUF_5e99fbf41e40[buf_idx_5e99fbf41e40 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5e99fbf581e0(uint64_t* BUF_5e99fbf23fe0, uint64_t* BUF_5e99fbf41e40, uint64_t* BUF_5e99fbf47bf0, uint64_t* COUNT5e99fbf472e0, HASHTABLE_PROBE HT_5e99fbf23fe0, HASHTABLE_PROBE HT_5e99fbf41e40, HASHTABLE_PROBE HT_5e99fbf47bf0, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e99fbf47bf0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5e99fbf47bf0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5e99fbf47bf0.for_each(KEY_5e99fbf47bf0, [&] __device__ (auto const SLOT_5e99fbf47bf0) {

auto const [slot_first5e99fbf47bf0, slot_second5e99fbf47bf0] = SLOT_5e99fbf47bf0;
if (!(true)) return;
uint64_t KEY_5e99fbf23fe0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5e99fbf23fe0 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5e99fbf23fe0.for_each(KEY_5e99fbf23fe0, [&] __device__ (auto const SLOT_5e99fbf23fe0) {

auto const [slot_first5e99fbf23fe0, slot_second5e99fbf23fe0] = SLOT_5e99fbf23fe0;
if (!(true)) return;
uint64_t KEY_5e99fbf41e40 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5e99fbf41e40 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5e99fbf41e40.for_each(KEY_5e99fbf41e40, [&] __device__ (auto const SLOT_5e99fbf41e40) {

auto const [slot_first5e99fbf41e40, slot_second5e99fbf41e40] = SLOT_5e99fbf41e40;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5e99fbf472e0, 1);
});
});
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5e99fbf581e0(uint64_t* BUF_5e99fbf23fe0, uint64_t* BUF_5e99fbf41e40, uint64_t* BUF_5e99fbf472e0, uint64_t* BUF_5e99fbf47bf0, uint64_t* BUF_IDX_5e99fbf472e0, HASHTABLE_PROBE HT_5e99fbf23fe0, HASHTABLE_PROBE HT_5e99fbf41e40, HASHTABLE_INSERT HT_5e99fbf472e0, HASHTABLE_PROBE HT_5e99fbf47bf0, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e99fbf47bf0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5e99fbf47bf0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5e99fbf47bf0.for_each(KEY_5e99fbf47bf0, [&] __device__ (auto const SLOT_5e99fbf47bf0) {
auto const [slot_first5e99fbf47bf0, slot_second5e99fbf47bf0] = SLOT_5e99fbf47bf0;
if (!(true)) return;
uint64_t KEY_5e99fbf23fe0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5e99fbf23fe0 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5e99fbf23fe0.for_each(KEY_5e99fbf23fe0, [&] __device__ (auto const SLOT_5e99fbf23fe0) {
auto const [slot_first5e99fbf23fe0, slot_second5e99fbf23fe0] = SLOT_5e99fbf23fe0;
if (!(true)) return;
uint64_t KEY_5e99fbf41e40 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5e99fbf41e40 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5e99fbf41e40.for_each(KEY_5e99fbf41e40, [&] __device__ (auto const SLOT_5e99fbf41e40) {
auto const [slot_first5e99fbf41e40, slot_second5e99fbf41e40] = SLOT_5e99fbf41e40;
if (!(true)) return;
uint64_t KEY_5e99fbf472e0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5e99fbf472e0 |= reg_lineorder__lo_custkey;
// Insert hash table kernel;
auto buf_idx_5e99fbf472e0 = atomicAdd((int*)BUF_IDX_5e99fbf472e0, 1);
HT_5e99fbf472e0.insert(cuco::pair{KEY_5e99fbf472e0, buf_idx_5e99fbf472e0});
BUF_5e99fbf472e0[buf_idx_5e99fbf472e0 * 4 + 0] = BUF_5e99fbf41e40[slot_second5e99fbf41e40 * 1 + 0];
BUF_5e99fbf472e0[buf_idx_5e99fbf472e0 * 4 + 1] = BUF_5e99fbf47bf0[slot_second5e99fbf47bf0 * 1 + 0];
BUF_5e99fbf472e0[buf_idx_5e99fbf472e0 * 4 + 2] = BUF_5e99fbf23fe0[slot_second5e99fbf23fe0 * 1 + 0];
BUF_5e99fbf472e0[buf_idx_5e99fbf472e0 * 4 + 3] = tid;
});
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5e99fbeead10(uint64_t* BUF_5e99fbf472e0, HASHTABLE_INSERT HT_5e99fbef96f0, HASHTABLE_PROBE HT_5e99fbf472e0, DBI32Type* customer__c_custkey, size_t customer_size, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e99fbf472e0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5e99fbf472e0 |= reg_customer__c_custkey;
//Probe Hash table
HT_5e99fbf472e0.for_each(KEY_5e99fbf472e0, [&] __device__ (auto const SLOT_5e99fbf472e0) {

auto const [slot_first5e99fbf472e0, slot_second5e99fbf472e0] = SLOT_5e99fbf472e0;
if (!(true)) return;
uint64_t KEY_5e99fbef96f0 = 0;
auto reg_date__d_year = date__d_year[BUF_5e99fbf472e0[slot_second5e99fbf472e0 * 4 + 0]];

KEY_5e99fbef96f0 |= reg_date__d_year;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_5e99fbf472e0[slot_second5e99fbf472e0 * 4 + 1]];
KEY_5e99fbef96f0 <<= 16;
KEY_5e99fbef96f0 |= reg_supplier__s_city_encoded;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_5e99fbf472e0[slot_second5e99fbf472e0 * 4 + 2]];
KEY_5e99fbef96f0 <<= 16;
KEY_5e99fbef96f0 |= reg_part__p_brand1_encoded;
//Create aggregation hash table
HT_5e99fbef96f0.insert(cuco::pair{KEY_5e99fbef96f0, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5e99fbeead10(uint64_t* BUF_5e99fbf472e0, HASHTABLE_FIND HT_5e99fbef96f0, HASHTABLE_PROBE HT_5e99fbf472e0, DBI32Type* KEY_5e99fbef96f0date__d_year, DBI16Type* KEY_5e99fbef96f0part__p_brand1_encoded, DBI16Type* KEY_5e99fbef96f0supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* customer__c_custkey, size_t customer_size, DBI32Type* date__d_year, DBDecimalType* lineorder__lo_revenue, DBDecimalType* lineorder__lo_supplycost, DBI16Type* part__p_brand1_encoded, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e99fbf472e0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5e99fbf472e0 |= reg_customer__c_custkey;
//Probe Hash table
HT_5e99fbf472e0.for_each(KEY_5e99fbf472e0, [&] __device__ (auto const SLOT_5e99fbf472e0) {
auto const [slot_first5e99fbf472e0, slot_second5e99fbf472e0] = SLOT_5e99fbf472e0;
if (!(true)) return;
uint64_t KEY_5e99fbef96f0 = 0;
auto reg_date__d_year = date__d_year[BUF_5e99fbf472e0[slot_second5e99fbf472e0 * 4 + 0]];

KEY_5e99fbef96f0 |= reg_date__d_year;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_5e99fbf472e0[slot_second5e99fbf472e0 * 4 + 1]];
KEY_5e99fbef96f0 <<= 16;
KEY_5e99fbef96f0 |= reg_supplier__s_city_encoded;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_5e99fbf472e0[slot_second5e99fbf472e0 * 4 + 2]];
KEY_5e99fbef96f0 <<= 16;
KEY_5e99fbef96f0 |= reg_part__p_brand1_encoded;
//Aggregate in hashtable
auto buf_idx_5e99fbef96f0 = HT_5e99fbef96f0.find(KEY_5e99fbef96f0)->second;
auto reg_lineorder__lo_supplycost = lineorder__lo_supplycost[BUF_5e99fbf472e0[slot_second5e99fbf472e0 * 4 + 3]];
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_5e99fbf472e0[slot_second5e99fbf472e0 * 4 + 3]];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_revenue) - (reg_lineorder__lo_supplycost);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5e99fbef96f0], reg_map0__tmp_attr1);
KEY_5e99fbef96f0date__d_year[buf_idx_5e99fbef96f0] = reg_date__d_year;
KEY_5e99fbef96f0supplier__s_city_encoded[buf_idx_5e99fbef96f0] = reg_supplier__s_city_encoded;
KEY_5e99fbef96f0part__p_brand1_encoded[buf_idx_5e99fbef96f0] = reg_part__p_brand1_encoded;
});
}
__global__ void count_5e99fbf70940(uint64_t* COUNT5e99fbed91b0, size_t COUNT5e99fbef96f0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5e99fbef96f0) return;
//Materialize count
atomicAdd((int*)COUNT5e99fbed91b0, 1);
}
__global__ void main_5e99fbf70940(size_t COUNT5e99fbef96f0, DBDecimalType* MAT5e99fbed91b0aggr0__tmp_attr0, DBI32Type* MAT5e99fbed91b0date__d_year, DBI16Type* MAT5e99fbed91b0part__p_brand1_encoded, DBI16Type* MAT5e99fbed91b0supplier__s_city_encoded, uint64_t* MAT_IDX5e99fbed91b0, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5e99fbef96f0) return;
//Materialize buffers
auto mat_idx5e99fbed91b0 = atomicAdd((int*)MAT_IDX5e99fbed91b0, 1);
auto reg_date__d_year = date__d_year[tid];
MAT5e99fbed91b0date__d_year[mat_idx5e99fbed91b0] = reg_date__d_year;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[tid];
MAT5e99fbed91b0supplier__s_city_encoded[mat_idx5e99fbed91b0] = reg_supplier__s_city_encoded;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[tid];
MAT5e99fbed91b0part__p_brand1_encoded[mat_idx5e99fbed91b0] = reg_part__p_brand1_encoded;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5e99fbed91b0aggr0__tmp_attr0[mat_idx5e99fbed91b0] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT5e99fbf47bf0;
hipMalloc(&d_COUNT5e99fbf47bf0, sizeof(uint64_t));
hipMemset(d_COUNT5e99fbf47bf0, 0, sizeof(uint64_t));
count_5e99fbeeaad0<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT5e99fbf47bf0, d_supplier__s_nation, supplier_size);
uint64_t COUNT5e99fbf47bf0;
hipMemcpy(&COUNT5e99fbf47bf0, d_COUNT5e99fbf47bf0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5e99fbf47bf0;
hipMalloc(&d_BUF_IDX_5e99fbf47bf0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5e99fbf47bf0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5e99fbf47bf0;
hipMalloc(&d_BUF_5e99fbf47bf0, sizeof(uint64_t) * COUNT5e99fbf47bf0 * 1);
auto d_HT_5e99fbf47bf0 = cuco::experimental::static_multimap{ (int)COUNT5e99fbf47bf0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5e99fbeeaad0<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_5e99fbf47bf0, d_BUF_IDX_5e99fbf47bf0, d_HT_5e99fbf47bf0.ref(cuco::insert), d_supplier__s_nation, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5e99fbf23fe0;
hipMalloc(&d_COUNT5e99fbf23fe0, sizeof(uint64_t));
hipMemset(d_COUNT5e99fbf23fe0, 0, sizeof(uint64_t));
count_5e99fbf55950<<<std::ceil((float)part_size/128.), 128>>>(d_COUNT5e99fbf23fe0, d_part__p_category, part_size);
uint64_t COUNT5e99fbf23fe0;
hipMemcpy(&COUNT5e99fbf23fe0, d_COUNT5e99fbf23fe0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5e99fbf23fe0;
hipMalloc(&d_BUF_IDX_5e99fbf23fe0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5e99fbf23fe0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5e99fbf23fe0;
hipMalloc(&d_BUF_5e99fbf23fe0, sizeof(uint64_t) * COUNT5e99fbf23fe0 * 1);
auto d_HT_5e99fbf23fe0 = cuco::experimental::static_multimap{ (int)COUNT5e99fbf23fe0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5e99fbf55950<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_5e99fbf23fe0, d_BUF_IDX_5e99fbf23fe0, d_HT_5e99fbf23fe0.ref(cuco::insert), d_part__p_category, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT5e99fbf41e40;
hipMalloc(&d_COUNT5e99fbf41e40, sizeof(uint64_t));
hipMemset(d_COUNT5e99fbf41e40, 0, sizeof(uint64_t));
count_5e99fbf450d0<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT5e99fbf41e40, d_date__d_year, date_size);
uint64_t COUNT5e99fbf41e40;
hipMemcpy(&COUNT5e99fbf41e40, d_COUNT5e99fbf41e40, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5e99fbf41e40;
hipMalloc(&d_BUF_IDX_5e99fbf41e40, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5e99fbf41e40, 0, sizeof(uint64_t));
uint64_t* d_BUF_5e99fbf41e40;
hipMalloc(&d_BUF_5e99fbf41e40, sizeof(uint64_t) * COUNT5e99fbf41e40 * 1);
auto d_HT_5e99fbf41e40 = cuco::experimental::static_multimap{ (int)COUNT5e99fbf41e40*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5e99fbf450d0<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_5e99fbf41e40, d_BUF_IDX_5e99fbf41e40, d_HT_5e99fbf41e40.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Materialize count
uint64_t* d_COUNT5e99fbf472e0;
hipMalloc(&d_COUNT5e99fbf472e0, sizeof(uint64_t));
hipMemset(d_COUNT5e99fbf472e0, 0, sizeof(uint64_t));
count_5e99fbf581e0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5e99fbf23fe0, d_BUF_5e99fbf41e40, d_BUF_5e99fbf47bf0, d_COUNT5e99fbf472e0, d_HT_5e99fbf23fe0.ref(cuco::for_each), d_HT_5e99fbf41e40.ref(cuco::for_each), d_HT_5e99fbf47bf0.ref(cuco::for_each), d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
uint64_t COUNT5e99fbf472e0;
hipMemcpy(&COUNT5e99fbf472e0, d_COUNT5e99fbf472e0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5e99fbf472e0;
hipMalloc(&d_BUF_IDX_5e99fbf472e0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5e99fbf472e0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5e99fbf472e0;
hipMalloc(&d_BUF_5e99fbf472e0, sizeof(uint64_t) * COUNT5e99fbf472e0 * 4);
auto d_HT_5e99fbf472e0 = cuco::experimental::static_multimap{ (int)COUNT5e99fbf472e0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5e99fbf581e0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5e99fbf23fe0, d_BUF_5e99fbf41e40, d_BUF_5e99fbf472e0, d_BUF_5e99fbf47bf0, d_BUF_IDX_5e99fbf472e0, d_HT_5e99fbf23fe0.ref(cuco::for_each), d_HT_5e99fbf41e40.ref(cuco::for_each), d_HT_5e99fbf472e0.ref(cuco::insert), d_HT_5e99fbf47bf0.ref(cuco::for_each), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
//Create aggregation hash table
auto d_HT_5e99fbef96f0 = cuco::static_map{ (int)2259*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5e99fbeead10<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_5e99fbf472e0, d_HT_5e99fbef96f0.ref(cuco::insert), d_HT_5e99fbf472e0.ref(cuco::for_each), d_customer__c_custkey, customer_size, d_date__d_year, d_part__p_brand1_encoded, d_supplier__s_city_encoded);
size_t COUNT5e99fbef96f0 = d_HT_5e99fbef96f0.size();
thrust::device_vector<int64_t> keys_5e99fbef96f0(COUNT5e99fbef96f0), vals_5e99fbef96f0(COUNT5e99fbef96f0);
d_HT_5e99fbef96f0.retrieve_all(keys_5e99fbef96f0.begin(), vals_5e99fbef96f0.begin());
d_HT_5e99fbef96f0.clear();
int64_t* raw_keys5e99fbef96f0 = thrust::raw_pointer_cast(keys_5e99fbef96f0.data());
insertKeys<<<std::ceil((float)COUNT5e99fbef96f0/128.), 128>>>(raw_keys5e99fbef96f0, d_HT_5e99fbef96f0.ref(cuco::insert), COUNT5e99fbef96f0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5e99fbef96f0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5e99fbef96f0);
DBI32Type* d_KEY_5e99fbef96f0date__d_year;
hipMalloc(&d_KEY_5e99fbef96f0date__d_year, sizeof(DBI32Type) * COUNT5e99fbef96f0);
hipMemset(d_KEY_5e99fbef96f0date__d_year, 0, sizeof(DBI32Type) * COUNT5e99fbef96f0);
DBI16Type* d_KEY_5e99fbef96f0supplier__s_city_encoded;
hipMalloc(&d_KEY_5e99fbef96f0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5e99fbef96f0);
hipMemset(d_KEY_5e99fbef96f0supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT5e99fbef96f0);
DBI16Type* d_KEY_5e99fbef96f0part__p_brand1_encoded;
hipMalloc(&d_KEY_5e99fbef96f0part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5e99fbef96f0);
hipMemset(d_KEY_5e99fbef96f0part__p_brand1_encoded, 0, sizeof(DBI16Type) * COUNT5e99fbef96f0);
main_5e99fbeead10<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_5e99fbf472e0, d_HT_5e99fbef96f0.ref(cuco::find), d_HT_5e99fbf472e0.ref(cuco::for_each), d_KEY_5e99fbef96f0date__d_year, d_KEY_5e99fbef96f0part__p_brand1_encoded, d_KEY_5e99fbef96f0supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_custkey, customer_size, d_date__d_year, d_lineorder__lo_revenue, d_lineorder__lo_supplycost, d_part__p_brand1_encoded, d_supplier__s_city_encoded);
//Materialize count
uint64_t* d_COUNT5e99fbed91b0;
hipMalloc(&d_COUNT5e99fbed91b0, sizeof(uint64_t));
hipMemset(d_COUNT5e99fbed91b0, 0, sizeof(uint64_t));
count_5e99fbf70940<<<std::ceil((float)COUNT5e99fbef96f0/128.), 128>>>(d_COUNT5e99fbed91b0, COUNT5e99fbef96f0);
uint64_t COUNT5e99fbed91b0;
hipMemcpy(&COUNT5e99fbed91b0, d_COUNT5e99fbed91b0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5e99fbed91b0;
hipMalloc(&d_MAT_IDX5e99fbed91b0, sizeof(uint64_t));
hipMemset(d_MAT_IDX5e99fbed91b0, 0, sizeof(uint64_t));
auto MAT5e99fbed91b0date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5e99fbed91b0);
DBI32Type* d_MAT5e99fbed91b0date__d_year;
hipMalloc(&d_MAT5e99fbed91b0date__d_year, sizeof(DBI32Type) * COUNT5e99fbed91b0);
auto MAT5e99fbed91b0supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5e99fbed91b0);
DBI16Type* d_MAT5e99fbed91b0supplier__s_city_encoded;
hipMalloc(&d_MAT5e99fbed91b0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5e99fbed91b0);
auto MAT5e99fbed91b0part__p_brand1_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5e99fbed91b0);
DBI16Type* d_MAT5e99fbed91b0part__p_brand1_encoded;
hipMalloc(&d_MAT5e99fbed91b0part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5e99fbed91b0);
auto MAT5e99fbed91b0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5e99fbed91b0);
DBDecimalType* d_MAT5e99fbed91b0aggr0__tmp_attr0;
hipMalloc(&d_MAT5e99fbed91b0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5e99fbed91b0);
main_5e99fbf70940<<<std::ceil((float)COUNT5e99fbef96f0/128.), 128>>>(COUNT5e99fbef96f0, d_MAT5e99fbed91b0aggr0__tmp_attr0, d_MAT5e99fbed91b0date__d_year, d_MAT5e99fbed91b0part__p_brand1_encoded, d_MAT5e99fbed91b0supplier__s_city_encoded, d_MAT_IDX5e99fbed91b0, d_aggr0__tmp_attr0, d_KEY_5e99fbef96f0date__d_year, d_KEY_5e99fbef96f0part__p_brand1_encoded, d_KEY_5e99fbef96f0supplier__s_city_encoded);
hipMemcpy(MAT5e99fbed91b0date__d_year, d_MAT5e99fbed91b0date__d_year, sizeof(DBI32Type) * COUNT5e99fbed91b0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5e99fbed91b0supplier__s_city_encoded, d_MAT5e99fbed91b0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5e99fbed91b0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5e99fbed91b0part__p_brand1_encoded, d_MAT5e99fbed91b0part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5e99fbed91b0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5e99fbed91b0aggr0__tmp_attr0, d_MAT5e99fbed91b0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5e99fbed91b0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5e99fbed91b0; i++) { std::cout << "" << MAT5e99fbed91b0date__d_year[i];
std::cout << "," << supplier__s_city_map[MAT5e99fbed91b0supplier__s_city_encoded[i]];
std::cout << "," << part__p_brand1_map[MAT5e99fbed91b0part__p_brand1_encoded[i]];
std::cout << "," << MAT5e99fbed91b0aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_5e99fbf47bf0);
hipFree(d_BUF_IDX_5e99fbf47bf0);
hipFree(d_COUNT5e99fbf47bf0);
hipFree(d_BUF_5e99fbf23fe0);
hipFree(d_BUF_IDX_5e99fbf23fe0);
hipFree(d_COUNT5e99fbf23fe0);
hipFree(d_BUF_5e99fbf41e40);
hipFree(d_BUF_IDX_5e99fbf41e40);
hipFree(d_COUNT5e99fbf41e40);
hipFree(d_BUF_5e99fbf472e0);
hipFree(d_BUF_IDX_5e99fbf472e0);
hipFree(d_COUNT5e99fbf472e0);
hipFree(d_KEY_5e99fbef96f0date__d_year);
hipFree(d_KEY_5e99fbef96f0part__p_brand1_encoded);
hipFree(d_KEY_5e99fbef96f0supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5e99fbed91b0);
hipFree(d_MAT5e99fbed91b0aggr0__tmp_attr0);
hipFree(d_MAT5e99fbed91b0date__d_year);
hipFree(d_MAT5e99fbed91b0part__p_brand1_encoded);
hipFree(d_MAT5e99fbed91b0supplier__s_city_encoded);
hipFree(d_MAT_IDX5e99fbed91b0);
free(MAT5e99fbed91b0aggr0__tmp_attr0);
free(MAT5e99fbed91b0date__d_year);
free(MAT5e99fbed91b0part__p_brand1_encoded);
free(MAT5e99fbed91b0supplier__s_city_encoded);
}