#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5999e9966490(uint64_t* COUNT5999e9959fb0, DBStringType* supplier__s_nation, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5999e9959fb0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5999e9966490(uint64_t* BUF_5999e9959fb0, uint64_t* BUF_IDX_5999e9959fb0, HASHTABLE_INSERT HT_5999e9959fb0, DBStringType* supplier__s_nation, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5999e9959fb0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5999e9959fb0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5999e9959fb0 = atomicAdd((int*)BUF_IDX_5999e9959fb0, 1);
HT_5999e9959fb0.insert(cuco::pair{KEY_5999e9959fb0, buf_idx_5999e9959fb0});
BUF_5999e9959fb0[buf_idx_5999e9959fb0 * 1 + 0] = tid;
}
__global__ void count_5999e9968e50(uint64_t* COUNT5999e995a920, DBStringType* part__p_category, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_category = part__p_category[tid];
if (!(evaluatePredicate(reg_part__p_category, "MFGR#14", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5999e995a920, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5999e9968e50(uint64_t* BUF_5999e995a920, uint64_t* BUF_IDX_5999e995a920, HASHTABLE_INSERT HT_5999e995a920, DBStringType* part__p_category, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_category = part__p_category[tid];
if (!(evaluatePredicate(reg_part__p_category, "MFGR#14", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5999e995a920 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_5999e995a920 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_5999e995a920 = atomicAdd((int*)BUF_IDX_5999e995a920, 1);
HT_5999e995a920.insert(cuco::pair{KEY_5999e995a920, buf_idx_5999e995a920});
BUF_5999e995a920[buf_idx_5999e995a920 * 1 + 0] = tid;
}
__global__ void count_5999e98ff9f0(uint64_t* COUNT5999e9956460, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5999e9956460, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5999e98ff9f0(uint64_t* BUF_5999e9956460, uint64_t* BUF_IDX_5999e9956460, HASHTABLE_INSERT HT_5999e9956460, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5999e9956460 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5999e9956460 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_5999e9956460 = atomicAdd((int*)BUF_IDX_5999e9956460, 1);
HT_5999e9956460.insert(cuco::pair{KEY_5999e9956460, buf_idx_5999e9956460});
BUF_5999e9956460[buf_idx_5999e9956460 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5999e996b750(uint64_t* BUF_5999e9956460, uint64_t* BUF_5999e9959fb0, uint64_t* BUF_5999e995a920, uint64_t* COUNT5999e9938850, HASHTABLE_PROBE HT_5999e9956460, HASHTABLE_PROBE HT_5999e9959fb0, HASHTABLE_PROBE HT_5999e995a920, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5999e9959fb0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5999e9959fb0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5999e9959fb0.for_each(KEY_5999e9959fb0, [&] __device__ (auto const SLOT_5999e9959fb0) {

auto const [slot_first5999e9959fb0, slot_second5999e9959fb0] = SLOT_5999e9959fb0;
if (!(true)) return;
uint64_t KEY_5999e995a920 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5999e995a920 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5999e995a920.for_each(KEY_5999e995a920, [&] __device__ (auto const SLOT_5999e995a920) {

auto const [slot_first5999e995a920, slot_second5999e995a920] = SLOT_5999e995a920;
if (!(true)) return;
uint64_t KEY_5999e9956460 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5999e9956460 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5999e9956460.for_each(KEY_5999e9956460, [&] __device__ (auto const SLOT_5999e9956460) {

auto const [slot_first5999e9956460, slot_second5999e9956460] = SLOT_5999e9956460;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5999e9938850, 1);
});
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_5999e996b750(uint64_t* BUF_5999e9938850, uint64_t* BUF_5999e9956460, uint64_t* BUF_5999e9959fb0, uint64_t* BUF_5999e995a920, uint64_t* BUF_IDX_5999e9938850, HASHTABLE_INSERT HT_5999e9938850, HASHTABLE_PROBE HT_5999e9956460, HASHTABLE_PROBE HT_5999e9959fb0, HASHTABLE_PROBE HT_5999e995a920, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5999e9959fb0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5999e9959fb0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5999e9959fb0.for_each(KEY_5999e9959fb0, [&] __device__ (auto const SLOT_5999e9959fb0) {
auto const [slot_first5999e9959fb0, slot_second5999e9959fb0] = SLOT_5999e9959fb0;
if (!(true)) return;
uint64_t KEY_5999e995a920 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5999e995a920 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5999e995a920.for_each(KEY_5999e995a920, [&] __device__ (auto const SLOT_5999e995a920) {
auto const [slot_first5999e995a920, slot_second5999e995a920] = SLOT_5999e995a920;
if (!(true)) return;
uint64_t KEY_5999e9956460 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5999e9956460 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5999e9956460.for_each(KEY_5999e9956460, [&] __device__ (auto const SLOT_5999e9956460) {
auto const [slot_first5999e9956460, slot_second5999e9956460] = SLOT_5999e9956460;
if (!(true)) return;
uint64_t KEY_5999e9938850 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5999e9938850 |= reg_lineorder__lo_custkey;
// Insert hash table kernel;
auto buf_idx_5999e9938850 = atomicAdd((int*)BUF_IDX_5999e9938850, 1);
HT_5999e9938850.insert(cuco::pair{KEY_5999e9938850, buf_idx_5999e9938850});
BUF_5999e9938850[buf_idx_5999e9938850 * 4 + 0] = BUF_5999e9956460[slot_second5999e9956460 * 1 + 0];
BUF_5999e9938850[buf_idx_5999e9938850 * 4 + 1] = BUF_5999e9959fb0[slot_second5999e9959fb0 * 1 + 0];
BUF_5999e9938850[buf_idx_5999e9938850 * 4 + 2] = BUF_5999e995a920[slot_second5999e995a920 * 1 + 0];
BUF_5999e9938850[buf_idx_5999e9938850 * 4 + 3] = tid;
});
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5999e98ff7b0(uint64_t* BUF_5999e9938850, HASHTABLE_INSERT HT_5999e990db50, HASHTABLE_PROBE HT_5999e9938850, DBI32Type* customer__c_custkey, size_t customer_size, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5999e9938850 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5999e9938850 |= reg_customer__c_custkey;
//Probe Hash table
HT_5999e9938850.for_each(KEY_5999e9938850, [&] __device__ (auto const SLOT_5999e9938850) {

auto const [slot_first5999e9938850, slot_second5999e9938850] = SLOT_5999e9938850;
if (!(true)) return;
uint64_t KEY_5999e990db50 = 0;
auto reg_date__d_year = date__d_year[BUF_5999e9938850[slot_second5999e9938850 * 4 + 0]];

KEY_5999e990db50 |= reg_date__d_year;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_5999e9938850[slot_second5999e9938850 * 4 + 1]];
KEY_5999e990db50 <<= 16;
KEY_5999e990db50 |= reg_supplier__s_city_encoded;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_5999e9938850[slot_second5999e9938850 * 4 + 2]];
KEY_5999e990db50 <<= 16;
KEY_5999e990db50 |= reg_part__p_brand1_encoded;
//Create aggregation hash table
HT_5999e990db50.insert(cuco::pair{KEY_5999e990db50, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5999e98ff7b0(uint64_t* BUF_5999e9938850, HASHTABLE_FIND HT_5999e990db50, HASHTABLE_PROBE HT_5999e9938850, DBI32Type* KEY_5999e990db50date__d_year, DBI16Type* KEY_5999e990db50part__p_brand1_encoded, DBI16Type* KEY_5999e990db50supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* customer__c_custkey, size_t customer_size, DBI32Type* date__d_year, DBDecimalType* lineorder__lo_revenue, DBDecimalType* lineorder__lo_supplycost, DBI16Type* part__p_brand1_encoded, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5999e9938850 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5999e9938850 |= reg_customer__c_custkey;
//Probe Hash table
HT_5999e9938850.for_each(KEY_5999e9938850, [&] __device__ (auto const SLOT_5999e9938850) {
auto const [slot_first5999e9938850, slot_second5999e9938850] = SLOT_5999e9938850;
if (!(true)) return;
uint64_t KEY_5999e990db50 = 0;
auto reg_date__d_year = date__d_year[BUF_5999e9938850[slot_second5999e9938850 * 4 + 0]];

KEY_5999e990db50 |= reg_date__d_year;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_5999e9938850[slot_second5999e9938850 * 4 + 1]];
KEY_5999e990db50 <<= 16;
KEY_5999e990db50 |= reg_supplier__s_city_encoded;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_5999e9938850[slot_second5999e9938850 * 4 + 2]];
KEY_5999e990db50 <<= 16;
KEY_5999e990db50 |= reg_part__p_brand1_encoded;
//Aggregate in hashtable
auto buf_idx_5999e990db50 = HT_5999e990db50.find(KEY_5999e990db50)->second;
auto reg_lineorder__lo_supplycost = lineorder__lo_supplycost[BUF_5999e9938850[slot_second5999e9938850 * 4 + 3]];
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_5999e9938850[slot_second5999e9938850 * 4 + 3]];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_revenue) - (reg_lineorder__lo_supplycost);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5999e990db50], reg_map0__tmp_attr1);
KEY_5999e990db50date__d_year[buf_idx_5999e990db50] = reg_date__d_year;
KEY_5999e990db50supplier__s_city_encoded[buf_idx_5999e990db50] = reg_supplier__s_city_encoded;
KEY_5999e990db50part__p_brand1_encoded[buf_idx_5999e990db50] = reg_part__p_brand1_encoded;
});
}
__global__ void count_5999e9984120(uint64_t* COUNT5999e98ede30, size_t COUNT5999e990db50) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5999e990db50) return;
//Materialize count
atomicAdd((int*)COUNT5999e98ede30, 1);
}
__global__ void main_5999e9984120(size_t COUNT5999e990db50, DBDecimalType* MAT5999e98ede30aggr0__tmp_attr0, DBI32Type* MAT5999e98ede30date__d_year, DBI16Type* MAT5999e98ede30part__p_brand1_encoded, DBI16Type* MAT5999e98ede30supplier__s_city_encoded, uint64_t* MAT_IDX5999e98ede30, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5999e990db50) return;
//Materialize buffers
auto mat_idx5999e98ede30 = atomicAdd((int*)MAT_IDX5999e98ede30, 1);
auto reg_date__d_year = date__d_year[tid];
MAT5999e98ede30date__d_year[mat_idx5999e98ede30] = reg_date__d_year;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[tid];
MAT5999e98ede30supplier__s_city_encoded[mat_idx5999e98ede30] = reg_supplier__s_city_encoded;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[tid];
MAT5999e98ede30part__p_brand1_encoded[mat_idx5999e98ede30] = reg_part__p_brand1_encoded;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5999e98ede30aggr0__tmp_attr0[mat_idx5999e98ede30] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT5999e9959fb0;
hipMalloc(&d_COUNT5999e9959fb0, sizeof(uint64_t));
hipMemset(d_COUNT5999e9959fb0, 0, sizeof(uint64_t));
count_5999e9966490<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT5999e9959fb0, d_supplier__s_nation, supplier_size);
uint64_t COUNT5999e9959fb0;
hipMemcpy(&COUNT5999e9959fb0, d_COUNT5999e9959fb0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5999e9959fb0;
hipMalloc(&d_BUF_IDX_5999e9959fb0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5999e9959fb0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5999e9959fb0;
hipMalloc(&d_BUF_5999e9959fb0, sizeof(uint64_t) * COUNT5999e9959fb0 * 1);
auto d_HT_5999e9959fb0 = cuco::experimental::static_multimap{ (int)COUNT5999e9959fb0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5999e9966490<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_5999e9959fb0, d_BUF_IDX_5999e9959fb0, d_HT_5999e9959fb0.ref(cuco::insert), d_supplier__s_nation, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5999e995a920;
hipMalloc(&d_COUNT5999e995a920, sizeof(uint64_t));
hipMemset(d_COUNT5999e995a920, 0, sizeof(uint64_t));
count_5999e9968e50<<<std::ceil((float)part_size/32.), 32>>>(d_COUNT5999e995a920, d_part__p_category, part_size);
uint64_t COUNT5999e995a920;
hipMemcpy(&COUNT5999e995a920, d_COUNT5999e995a920, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5999e995a920;
hipMalloc(&d_BUF_IDX_5999e995a920, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5999e995a920, 0, sizeof(uint64_t));
uint64_t* d_BUF_5999e995a920;
hipMalloc(&d_BUF_5999e995a920, sizeof(uint64_t) * COUNT5999e995a920 * 1);
auto d_HT_5999e995a920 = cuco::experimental::static_multimap{ (int)COUNT5999e995a920*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5999e9968e50<<<std::ceil((float)part_size/32.), 32>>>(d_BUF_5999e995a920, d_BUF_IDX_5999e995a920, d_HT_5999e995a920.ref(cuco::insert), d_part__p_category, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT5999e9956460;
hipMalloc(&d_COUNT5999e9956460, sizeof(uint64_t));
hipMemset(d_COUNT5999e9956460, 0, sizeof(uint64_t));
count_5999e98ff9f0<<<std::ceil((float)date_size/32.), 32>>>(d_COUNT5999e9956460, d_date__d_year, date_size);
uint64_t COUNT5999e9956460;
hipMemcpy(&COUNT5999e9956460, d_COUNT5999e9956460, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5999e9956460;
hipMalloc(&d_BUF_IDX_5999e9956460, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5999e9956460, 0, sizeof(uint64_t));
uint64_t* d_BUF_5999e9956460;
hipMalloc(&d_BUF_5999e9956460, sizeof(uint64_t) * COUNT5999e9956460 * 1);
auto d_HT_5999e9956460 = cuco::experimental::static_multimap{ (int)COUNT5999e9956460*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5999e98ff9f0<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_5999e9956460, d_BUF_IDX_5999e9956460, d_HT_5999e9956460.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Materialize count
uint64_t* d_COUNT5999e9938850;
hipMalloc(&d_COUNT5999e9938850, sizeof(uint64_t));
hipMemset(d_COUNT5999e9938850, 0, sizeof(uint64_t));
count_5999e996b750<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_5999e9956460, d_BUF_5999e9959fb0, d_BUF_5999e995a920, d_COUNT5999e9938850, d_HT_5999e9956460.ref(cuco::for_each), d_HT_5999e9959fb0.ref(cuco::for_each), d_HT_5999e995a920.ref(cuco::for_each), d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
uint64_t COUNT5999e9938850;
hipMemcpy(&COUNT5999e9938850, d_COUNT5999e9938850, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5999e9938850;
hipMalloc(&d_BUF_IDX_5999e9938850, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5999e9938850, 0, sizeof(uint64_t));
uint64_t* d_BUF_5999e9938850;
hipMalloc(&d_BUF_5999e9938850, sizeof(uint64_t) * COUNT5999e9938850 * 4);
auto d_HT_5999e9938850 = cuco::experimental::static_multimap{ (int)COUNT5999e9938850*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5999e996b750<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_5999e9938850, d_BUF_5999e9956460, d_BUF_5999e9959fb0, d_BUF_5999e995a920, d_BUF_IDX_5999e9938850, d_HT_5999e9938850.ref(cuco::insert), d_HT_5999e9956460.ref(cuco::for_each), d_HT_5999e9959fb0.ref(cuco::for_each), d_HT_5999e995a920.ref(cuco::for_each), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
//Create aggregation hash table
auto d_HT_5999e990db50 = cuco::static_map{ (int)2259*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5999e98ff7b0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5999e9938850, d_HT_5999e990db50.ref(cuco::insert), d_HT_5999e9938850.ref(cuco::for_each), d_customer__c_custkey, customer_size, d_date__d_year, d_part__p_brand1_encoded, d_supplier__s_city_encoded);
size_t COUNT5999e990db50 = d_HT_5999e990db50.size();
thrust::device_vector<int64_t> keys_5999e990db50(COUNT5999e990db50), vals_5999e990db50(COUNT5999e990db50);
d_HT_5999e990db50.retrieve_all(keys_5999e990db50.begin(), vals_5999e990db50.begin());
d_HT_5999e990db50.clear();
int64_t* raw_keys5999e990db50 = thrust::raw_pointer_cast(keys_5999e990db50.data());
insertKeys<<<std::ceil((float)COUNT5999e990db50/32.), 32>>>(raw_keys5999e990db50, d_HT_5999e990db50.ref(cuco::insert), COUNT5999e990db50);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5999e990db50);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5999e990db50);
DBI32Type* d_KEY_5999e990db50date__d_year;
hipMalloc(&d_KEY_5999e990db50date__d_year, sizeof(DBI32Type) * COUNT5999e990db50);
hipMemset(d_KEY_5999e990db50date__d_year, 0, sizeof(DBI32Type) * COUNT5999e990db50);
DBI16Type* d_KEY_5999e990db50supplier__s_city_encoded;
hipMalloc(&d_KEY_5999e990db50supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5999e990db50);
hipMemset(d_KEY_5999e990db50supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT5999e990db50);
DBI16Type* d_KEY_5999e990db50part__p_brand1_encoded;
hipMalloc(&d_KEY_5999e990db50part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5999e990db50);
hipMemset(d_KEY_5999e990db50part__p_brand1_encoded, 0, sizeof(DBI16Type) * COUNT5999e990db50);
main_5999e98ff7b0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5999e9938850, d_HT_5999e990db50.ref(cuco::find), d_HT_5999e9938850.ref(cuco::for_each), d_KEY_5999e990db50date__d_year, d_KEY_5999e990db50part__p_brand1_encoded, d_KEY_5999e990db50supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_custkey, customer_size, d_date__d_year, d_lineorder__lo_revenue, d_lineorder__lo_supplycost, d_part__p_brand1_encoded, d_supplier__s_city_encoded);
//Materialize count
uint64_t* d_COUNT5999e98ede30;
hipMalloc(&d_COUNT5999e98ede30, sizeof(uint64_t));
hipMemset(d_COUNT5999e98ede30, 0, sizeof(uint64_t));
count_5999e9984120<<<std::ceil((float)COUNT5999e990db50/32.), 32>>>(d_COUNT5999e98ede30, COUNT5999e990db50);
uint64_t COUNT5999e98ede30;
hipMemcpy(&COUNT5999e98ede30, d_COUNT5999e98ede30, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5999e98ede30;
hipMalloc(&d_MAT_IDX5999e98ede30, sizeof(uint64_t));
hipMemset(d_MAT_IDX5999e98ede30, 0, sizeof(uint64_t));
auto MAT5999e98ede30date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5999e98ede30);
DBI32Type* d_MAT5999e98ede30date__d_year;
hipMalloc(&d_MAT5999e98ede30date__d_year, sizeof(DBI32Type) * COUNT5999e98ede30);
auto MAT5999e98ede30supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5999e98ede30);
DBI16Type* d_MAT5999e98ede30supplier__s_city_encoded;
hipMalloc(&d_MAT5999e98ede30supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5999e98ede30);
auto MAT5999e98ede30part__p_brand1_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5999e98ede30);
DBI16Type* d_MAT5999e98ede30part__p_brand1_encoded;
hipMalloc(&d_MAT5999e98ede30part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5999e98ede30);
auto MAT5999e98ede30aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5999e98ede30);
DBDecimalType* d_MAT5999e98ede30aggr0__tmp_attr0;
hipMalloc(&d_MAT5999e98ede30aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5999e98ede30);
main_5999e9984120<<<std::ceil((float)COUNT5999e990db50/32.), 32>>>(COUNT5999e990db50, d_MAT5999e98ede30aggr0__tmp_attr0, d_MAT5999e98ede30date__d_year, d_MAT5999e98ede30part__p_brand1_encoded, d_MAT5999e98ede30supplier__s_city_encoded, d_MAT_IDX5999e98ede30, d_aggr0__tmp_attr0, d_KEY_5999e990db50date__d_year, d_KEY_5999e990db50part__p_brand1_encoded, d_KEY_5999e990db50supplier__s_city_encoded);
hipMemcpy(MAT5999e98ede30date__d_year, d_MAT5999e98ede30date__d_year, sizeof(DBI32Type) * COUNT5999e98ede30, hipMemcpyDeviceToHost);
hipMemcpy(MAT5999e98ede30supplier__s_city_encoded, d_MAT5999e98ede30supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5999e98ede30, hipMemcpyDeviceToHost);
hipMemcpy(MAT5999e98ede30part__p_brand1_encoded, d_MAT5999e98ede30part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5999e98ede30, hipMemcpyDeviceToHost);
hipMemcpy(MAT5999e98ede30aggr0__tmp_attr0, d_MAT5999e98ede30aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5999e98ede30, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5999e98ede30; i++) { std::cout << MAT5999e98ede30date__d_year[i] << "\t";
std::cout << supplier__s_city_map[MAT5999e98ede30supplier__s_city_encoded[i]] << "\t";
std::cout << part__p_brand1_map[MAT5999e98ede30part__p_brand1_encoded[i]] << "\t";
std::cout << MAT5999e98ede30aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_5999e9959fb0);
hipFree(d_BUF_IDX_5999e9959fb0);
hipFree(d_COUNT5999e9959fb0);
hipFree(d_BUF_5999e995a920);
hipFree(d_BUF_IDX_5999e995a920);
hipFree(d_COUNT5999e995a920);
hipFree(d_BUF_5999e9956460);
hipFree(d_BUF_IDX_5999e9956460);
hipFree(d_COUNT5999e9956460);
hipFree(d_BUF_5999e9938850);
hipFree(d_BUF_IDX_5999e9938850);
hipFree(d_COUNT5999e9938850);
hipFree(d_KEY_5999e990db50date__d_year);
hipFree(d_KEY_5999e990db50part__p_brand1_encoded);
hipFree(d_KEY_5999e990db50supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5999e98ede30);
hipFree(d_MAT5999e98ede30aggr0__tmp_attr0);
hipFree(d_MAT5999e98ede30date__d_year);
hipFree(d_MAT5999e98ede30part__p_brand1_encoded);
hipFree(d_MAT5999e98ede30supplier__s_city_encoded);
hipFree(d_MAT_IDX5999e98ede30);
free(MAT5999e98ede30aggr0__tmp_attr0);
free(MAT5999e98ede30date__d_year);
free(MAT5999e98ede30part__p_brand1_encoded);
free(MAT5999e98ede30supplier__s_city_encoded);
}