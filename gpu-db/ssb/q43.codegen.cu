#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5caca179c3c0(uint64_t* COUNT5caca178c9e0, DBStringType* supplier__s_nation, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5caca178c9e0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5caca179c3c0(uint64_t* BUF_5caca178c9e0, uint64_t* BUF_IDX_5caca178c9e0, HASHTABLE_INSERT HT_5caca178c9e0, DBStringType* supplier__s_nation, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5caca178c9e0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5caca178c9e0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5caca178c9e0 = atomicAdd((int*)BUF_IDX_5caca178c9e0, 1);
HT_5caca178c9e0.insert(cuco::pair{KEY_5caca178c9e0, buf_idx_5caca178c9e0});
BUF_5caca178c9e0[buf_idx_5caca178c9e0 * 1 + 0] = tid;
}
__global__ void count_5caca179e870(uint64_t* COUNT5caca176e730, DBStringType* part__p_category, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_category = part__p_category[tid];
if (!(evaluatePredicate(reg_part__p_category, "MFGR#14", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5caca176e730, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5caca179e870(uint64_t* BUF_5caca176e730, uint64_t* BUF_IDX_5caca176e730, HASHTABLE_INSERT HT_5caca176e730, DBStringType* part__p_category, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_category = part__p_category[tid];
if (!(evaluatePredicate(reg_part__p_category, "MFGR#14", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5caca176e730 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_5caca176e730 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_5caca176e730 = atomicAdd((int*)BUF_IDX_5caca176e730, 1);
HT_5caca176e730.insert(cuco::pair{KEY_5caca176e730, buf_idx_5caca176e730});
BUF_5caca176e730[buf_idx_5caca176e730 * 1 + 0] = tid;
}
__global__ void count_5caca1734b80(uint64_t* COUNT5caca178f780, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5caca178f780, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5caca1734b80(uint64_t* BUF_5caca178f780, uint64_t* BUF_IDX_5caca178f780, HASHTABLE_INSERT HT_5caca178f780, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5caca178f780 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5caca178f780 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_5caca178f780 = atomicAdd((int*)BUF_IDX_5caca178f780, 1);
HT_5caca178f780.insert(cuco::pair{KEY_5caca178f780, buf_idx_5caca178f780});
BUF_5caca178f780[buf_idx_5caca178f780 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5caca17a0bc0(uint64_t* BUF_5caca176e730, uint64_t* BUF_5caca178c9e0, uint64_t* BUF_5caca178f780, uint64_t* COUNT5caca1791330, HASHTABLE_PROBE HT_5caca176e730, HASHTABLE_PROBE HT_5caca178c9e0, HASHTABLE_PROBE HT_5caca178f780, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5caca178c9e0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5caca178c9e0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5caca178c9e0.for_each(KEY_5caca178c9e0, [&] __device__ (auto const SLOT_5caca178c9e0) {

auto const [slot_first5caca178c9e0, slot_second5caca178c9e0] = SLOT_5caca178c9e0;
if (!(true)) return;
uint64_t KEY_5caca176e730 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5caca176e730 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5caca176e730.for_each(KEY_5caca176e730, [&] __device__ (auto const SLOT_5caca176e730) {

auto const [slot_first5caca176e730, slot_second5caca176e730] = SLOT_5caca176e730;
if (!(true)) return;
uint64_t KEY_5caca178f780 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5caca178f780 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5caca178f780.for_each(KEY_5caca178f780, [&] __device__ (auto const SLOT_5caca178f780) {

auto const [slot_first5caca178f780, slot_second5caca178f780] = SLOT_5caca178f780;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5caca1791330, 1);
});
});
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5caca17a0bc0(uint64_t* BUF_5caca176e730, uint64_t* BUF_5caca178c9e0, uint64_t* BUF_5caca178f780, uint64_t* BUF_5caca1791330, uint64_t* BUF_IDX_5caca1791330, HASHTABLE_PROBE HT_5caca176e730, HASHTABLE_PROBE HT_5caca178c9e0, HASHTABLE_PROBE HT_5caca178f780, HASHTABLE_INSERT HT_5caca1791330, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5caca178c9e0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5caca178c9e0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5caca178c9e0.for_each(KEY_5caca178c9e0, [&] __device__ (auto const SLOT_5caca178c9e0) {
auto const [slot_first5caca178c9e0, slot_second5caca178c9e0] = SLOT_5caca178c9e0;
if (!(true)) return;
uint64_t KEY_5caca176e730 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5caca176e730 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5caca176e730.for_each(KEY_5caca176e730, [&] __device__ (auto const SLOT_5caca176e730) {
auto const [slot_first5caca176e730, slot_second5caca176e730] = SLOT_5caca176e730;
if (!(true)) return;
uint64_t KEY_5caca178f780 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5caca178f780 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5caca178f780.for_each(KEY_5caca178f780, [&] __device__ (auto const SLOT_5caca178f780) {
auto const [slot_first5caca178f780, slot_second5caca178f780] = SLOT_5caca178f780;
if (!(true)) return;
uint64_t KEY_5caca1791330 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5caca1791330 |= reg_lineorder__lo_custkey;
// Insert hash table kernel;
auto buf_idx_5caca1791330 = atomicAdd((int*)BUF_IDX_5caca1791330, 1);
HT_5caca1791330.insert(cuco::pair{KEY_5caca1791330, buf_idx_5caca1791330});
BUF_5caca1791330[buf_idx_5caca1791330 * 4 + 0] = BUF_5caca178f780[slot_second5caca178f780 * 1 + 0];
BUF_5caca1791330[buf_idx_5caca1791330 * 4 + 1] = BUF_5caca178c9e0[slot_second5caca178c9e0 * 1 + 0];
BUF_5caca1791330[buf_idx_5caca1791330 * 4 + 2] = BUF_5caca176e730[slot_second5caca176e730 * 1 + 0];
BUF_5caca1791330[buf_idx_5caca1791330 * 4 + 3] = tid;
});
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5caca1734940(uint64_t* BUF_5caca1791330, HASHTABLE_INSERT HT_5caca17441d0, HASHTABLE_PROBE HT_5caca1791330, DBI32Type* customer__c_custkey, size_t customer_size, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5caca1791330 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5caca1791330 |= reg_customer__c_custkey;
//Probe Hash table
HT_5caca1791330.for_each(KEY_5caca1791330, [&] __device__ (auto const SLOT_5caca1791330) {

auto const [slot_first5caca1791330, slot_second5caca1791330] = SLOT_5caca1791330;
if (!(true)) return;
uint64_t KEY_5caca17441d0 = 0;
auto reg_date__d_year = date__d_year[BUF_5caca1791330[slot_second5caca1791330 * 4 + 0]];

KEY_5caca17441d0 |= reg_date__d_year;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_5caca1791330[slot_second5caca1791330 * 4 + 1]];
KEY_5caca17441d0 <<= 16;
KEY_5caca17441d0 |= reg_supplier__s_city_encoded;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_5caca1791330[slot_second5caca1791330 * 4 + 2]];
KEY_5caca17441d0 <<= 16;
KEY_5caca17441d0 |= reg_part__p_brand1_encoded;
//Create aggregation hash table
HT_5caca17441d0.insert(cuco::pair{KEY_5caca17441d0, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5caca1734940(uint64_t* BUF_5caca1791330, HASHTABLE_FIND HT_5caca17441d0, HASHTABLE_PROBE HT_5caca1791330, DBI32Type* KEY_5caca17441d0date__d_year, DBI16Type* KEY_5caca17441d0part__p_brand1_encoded, DBI16Type* KEY_5caca17441d0supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* customer__c_custkey, size_t customer_size, DBI32Type* date__d_year, DBDecimalType* lineorder__lo_revenue, DBDecimalType* lineorder__lo_supplycost, DBI16Type* part__p_brand1_encoded, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5caca1791330 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5caca1791330 |= reg_customer__c_custkey;
//Probe Hash table
HT_5caca1791330.for_each(KEY_5caca1791330, [&] __device__ (auto const SLOT_5caca1791330) {
auto const [slot_first5caca1791330, slot_second5caca1791330] = SLOT_5caca1791330;
if (!(true)) return;
uint64_t KEY_5caca17441d0 = 0;
auto reg_date__d_year = date__d_year[BUF_5caca1791330[slot_second5caca1791330 * 4 + 0]];

KEY_5caca17441d0 |= reg_date__d_year;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_5caca1791330[slot_second5caca1791330 * 4 + 1]];
KEY_5caca17441d0 <<= 16;
KEY_5caca17441d0 |= reg_supplier__s_city_encoded;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_5caca1791330[slot_second5caca1791330 * 4 + 2]];
KEY_5caca17441d0 <<= 16;
KEY_5caca17441d0 |= reg_part__p_brand1_encoded;
//Aggregate in hashtable
auto buf_idx_5caca17441d0 = HT_5caca17441d0.find(KEY_5caca17441d0)->second;
auto reg_lineorder__lo_supplycost = lineorder__lo_supplycost[BUF_5caca1791330[slot_second5caca1791330 * 4 + 3]];
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_5caca1791330[slot_second5caca1791330 * 4 + 3]];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_revenue) - (reg_lineorder__lo_supplycost);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5caca17441d0], reg_map0__tmp_attr1);
KEY_5caca17441d0date__d_year[buf_idx_5caca17441d0] = reg_date__d_year;
KEY_5caca17441d0supplier__s_city_encoded[buf_idx_5caca17441d0] = reg_supplier__s_city_encoded;
KEY_5caca17441d0part__p_brand1_encoded[buf_idx_5caca17441d0] = reg_part__p_brand1_encoded;
});
}
__global__ void count_5caca17b8660(uint64_t* COUNT5caca1723900, size_t COUNT5caca17441d0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5caca17441d0) return;
//Materialize count
atomicAdd((int*)COUNT5caca1723900, 1);
}
__global__ void main_5caca17b8660(size_t COUNT5caca17441d0, DBDecimalType* MAT5caca1723900aggr0__tmp_attr0, DBI32Type* MAT5caca1723900date__d_year, DBI16Type* MAT5caca1723900part__p_brand1_encoded, DBI16Type* MAT5caca1723900supplier__s_city_encoded, uint64_t* MAT_IDX5caca1723900, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5caca17441d0) return;
//Materialize buffers
auto mat_idx5caca1723900 = atomicAdd((int*)MAT_IDX5caca1723900, 1);
auto reg_date__d_year = date__d_year[tid];
MAT5caca1723900date__d_year[mat_idx5caca1723900] = reg_date__d_year;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[tid];
MAT5caca1723900supplier__s_city_encoded[mat_idx5caca1723900] = reg_supplier__s_city_encoded;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[tid];
MAT5caca1723900part__p_brand1_encoded[mat_idx5caca1723900] = reg_part__p_brand1_encoded;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5caca1723900aggr0__tmp_attr0[mat_idx5caca1723900] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map) {
//Materialize count
uint64_t* d_COUNT5caca178c9e0;
hipMalloc(&d_COUNT5caca178c9e0, sizeof(uint64_t));
hipMemset(d_COUNT5caca178c9e0, 0, sizeof(uint64_t));
count_5caca179c3c0<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT5caca178c9e0, d_supplier__s_nation, supplier_size);
uint64_t COUNT5caca178c9e0;
hipMemcpy(&COUNT5caca178c9e0, d_COUNT5caca178c9e0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5caca178c9e0;
hipMalloc(&d_BUF_IDX_5caca178c9e0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5caca178c9e0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5caca178c9e0;
hipMalloc(&d_BUF_5caca178c9e0, sizeof(uint64_t) * COUNT5caca178c9e0 * 1);
auto d_HT_5caca178c9e0 = cuco::experimental::static_multimap{ (int)COUNT5caca178c9e0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5caca179c3c0<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_5caca178c9e0, d_BUF_IDX_5caca178c9e0, d_HT_5caca178c9e0.ref(cuco::insert), d_supplier__s_nation, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5caca176e730;
hipMalloc(&d_COUNT5caca176e730, sizeof(uint64_t));
hipMemset(d_COUNT5caca176e730, 0, sizeof(uint64_t));
count_5caca179e870<<<std::ceil((float)part_size/32.), 32>>>(d_COUNT5caca176e730, d_part__p_category, part_size);
uint64_t COUNT5caca176e730;
hipMemcpy(&COUNT5caca176e730, d_COUNT5caca176e730, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5caca176e730;
hipMalloc(&d_BUF_IDX_5caca176e730, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5caca176e730, 0, sizeof(uint64_t));
uint64_t* d_BUF_5caca176e730;
hipMalloc(&d_BUF_5caca176e730, sizeof(uint64_t) * COUNT5caca176e730 * 1);
auto d_HT_5caca176e730 = cuco::experimental::static_multimap{ (int)COUNT5caca176e730*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5caca179e870<<<std::ceil((float)part_size/32.), 32>>>(d_BUF_5caca176e730, d_BUF_IDX_5caca176e730, d_HT_5caca176e730.ref(cuco::insert), d_part__p_category, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT5caca178f780;
hipMalloc(&d_COUNT5caca178f780, sizeof(uint64_t));
hipMemset(d_COUNT5caca178f780, 0, sizeof(uint64_t));
count_5caca1734b80<<<std::ceil((float)date_size/32.), 32>>>(d_COUNT5caca178f780, d_date__d_year, date_size);
uint64_t COUNT5caca178f780;
hipMemcpy(&COUNT5caca178f780, d_COUNT5caca178f780, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5caca178f780;
hipMalloc(&d_BUF_IDX_5caca178f780, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5caca178f780, 0, sizeof(uint64_t));
uint64_t* d_BUF_5caca178f780;
hipMalloc(&d_BUF_5caca178f780, sizeof(uint64_t) * COUNT5caca178f780 * 1);
auto d_HT_5caca178f780 = cuco::experimental::static_multimap{ (int)COUNT5caca178f780*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5caca1734b80<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_5caca178f780, d_BUF_IDX_5caca178f780, d_HT_5caca178f780.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Materialize count
uint64_t* d_COUNT5caca1791330;
hipMalloc(&d_COUNT5caca1791330, sizeof(uint64_t));
hipMemset(d_COUNT5caca1791330, 0, sizeof(uint64_t));
count_5caca17a0bc0<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_5caca176e730, d_BUF_5caca178c9e0, d_BUF_5caca178f780, d_COUNT5caca1791330, d_HT_5caca176e730.ref(cuco::for_each), d_HT_5caca178c9e0.ref(cuco::for_each), d_HT_5caca178f780.ref(cuco::for_each), d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
uint64_t COUNT5caca1791330;
hipMemcpy(&COUNT5caca1791330, d_COUNT5caca1791330, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5caca1791330;
hipMalloc(&d_BUF_IDX_5caca1791330, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5caca1791330, 0, sizeof(uint64_t));
uint64_t* d_BUF_5caca1791330;
hipMalloc(&d_BUF_5caca1791330, sizeof(uint64_t) * COUNT5caca1791330 * 4);
auto d_HT_5caca1791330 = cuco::experimental::static_multimap{ (int)COUNT5caca1791330*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5caca17a0bc0<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_5caca176e730, d_BUF_5caca178c9e0, d_BUF_5caca178f780, d_BUF_5caca1791330, d_BUF_IDX_5caca1791330, d_HT_5caca176e730.ref(cuco::for_each), d_HT_5caca178c9e0.ref(cuco::for_each), d_HT_5caca178f780.ref(cuco::for_each), d_HT_5caca1791330.ref(cuco::insert), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
//Create aggregation hash table
auto d_HT_5caca17441d0 = cuco::static_map{ (int)2259*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5caca1734940<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5caca1791330, d_HT_5caca17441d0.ref(cuco::insert), d_HT_5caca1791330.ref(cuco::for_each), d_customer__c_custkey, customer_size, d_date__d_year, d_part__p_brand1_encoded, d_supplier__s_city_encoded);
size_t COUNT5caca17441d0 = d_HT_5caca17441d0.size();
thrust::device_vector<int64_t> keys_5caca17441d0(COUNT5caca17441d0), vals_5caca17441d0(COUNT5caca17441d0);
d_HT_5caca17441d0.retrieve_all(keys_5caca17441d0.begin(), vals_5caca17441d0.begin());
d_HT_5caca17441d0.clear();
int64_t* raw_keys5caca17441d0 = thrust::raw_pointer_cast(keys_5caca17441d0.data());
insertKeys<<<std::ceil((float)COUNT5caca17441d0/32.), 32>>>(raw_keys5caca17441d0, d_HT_5caca17441d0.ref(cuco::insert), COUNT5caca17441d0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5caca17441d0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5caca17441d0);
DBI32Type* d_KEY_5caca17441d0date__d_year;
hipMalloc(&d_KEY_5caca17441d0date__d_year, sizeof(DBI32Type) * COUNT5caca17441d0);
hipMemset(d_KEY_5caca17441d0date__d_year, 0, sizeof(DBI32Type) * COUNT5caca17441d0);
DBI16Type* d_KEY_5caca17441d0supplier__s_city_encoded;
hipMalloc(&d_KEY_5caca17441d0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5caca17441d0);
hipMemset(d_KEY_5caca17441d0supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT5caca17441d0);
DBI16Type* d_KEY_5caca17441d0part__p_brand1_encoded;
hipMalloc(&d_KEY_5caca17441d0part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5caca17441d0);
hipMemset(d_KEY_5caca17441d0part__p_brand1_encoded, 0, sizeof(DBI16Type) * COUNT5caca17441d0);
main_5caca1734940<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5caca1791330, d_HT_5caca17441d0.ref(cuco::find), d_HT_5caca1791330.ref(cuco::for_each), d_KEY_5caca17441d0date__d_year, d_KEY_5caca17441d0part__p_brand1_encoded, d_KEY_5caca17441d0supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_custkey, customer_size, d_date__d_year, d_lineorder__lo_revenue, d_lineorder__lo_supplycost, d_part__p_brand1_encoded, d_supplier__s_city_encoded);
//Materialize count
uint64_t* d_COUNT5caca1723900;
hipMalloc(&d_COUNT5caca1723900, sizeof(uint64_t));
hipMemset(d_COUNT5caca1723900, 0, sizeof(uint64_t));
count_5caca17b8660<<<std::ceil((float)COUNT5caca17441d0/32.), 32>>>(d_COUNT5caca1723900, COUNT5caca17441d0);
uint64_t COUNT5caca1723900;
hipMemcpy(&COUNT5caca1723900, d_COUNT5caca1723900, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5caca1723900;
hipMalloc(&d_MAT_IDX5caca1723900, sizeof(uint64_t));
hipMemset(d_MAT_IDX5caca1723900, 0, sizeof(uint64_t));
auto MAT5caca1723900date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5caca1723900);
DBI32Type* d_MAT5caca1723900date__d_year;
hipMalloc(&d_MAT5caca1723900date__d_year, sizeof(DBI32Type) * COUNT5caca1723900);
auto MAT5caca1723900supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5caca1723900);
DBI16Type* d_MAT5caca1723900supplier__s_city_encoded;
hipMalloc(&d_MAT5caca1723900supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5caca1723900);
auto MAT5caca1723900part__p_brand1_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5caca1723900);
DBI16Type* d_MAT5caca1723900part__p_brand1_encoded;
hipMalloc(&d_MAT5caca1723900part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5caca1723900);
auto MAT5caca1723900aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5caca1723900);
DBDecimalType* d_MAT5caca1723900aggr0__tmp_attr0;
hipMalloc(&d_MAT5caca1723900aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5caca1723900);
main_5caca17b8660<<<std::ceil((float)COUNT5caca17441d0/32.), 32>>>(COUNT5caca17441d0, d_MAT5caca1723900aggr0__tmp_attr0, d_MAT5caca1723900date__d_year, d_MAT5caca1723900part__p_brand1_encoded, d_MAT5caca1723900supplier__s_city_encoded, d_MAT_IDX5caca1723900, d_aggr0__tmp_attr0, d_KEY_5caca17441d0date__d_year, d_KEY_5caca17441d0part__p_brand1_encoded, d_KEY_5caca17441d0supplier__s_city_encoded);
hipMemcpy(MAT5caca1723900date__d_year, d_MAT5caca1723900date__d_year, sizeof(DBI32Type) * COUNT5caca1723900, hipMemcpyDeviceToHost);
hipMemcpy(MAT5caca1723900supplier__s_city_encoded, d_MAT5caca1723900supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5caca1723900, hipMemcpyDeviceToHost);
hipMemcpy(MAT5caca1723900part__p_brand1_encoded, d_MAT5caca1723900part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5caca1723900, hipMemcpyDeviceToHost);
hipMemcpy(MAT5caca1723900aggr0__tmp_attr0, d_MAT5caca1723900aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5caca1723900, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5caca1723900; i++) { std::cout << MAT5caca1723900date__d_year[i] << "\t";
std::cout << supplier__s_city_map[MAT5caca1723900supplier__s_city_encoded[i]] << "\t";
std::cout << part__p_brand1_map[MAT5caca1723900part__p_brand1_encoded[i]] << "\t";
std::cout << MAT5caca1723900aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_5caca178c9e0);
hipFree(d_BUF_IDX_5caca178c9e0);
hipFree(d_COUNT5caca178c9e0);
hipFree(d_BUF_5caca176e730);
hipFree(d_BUF_IDX_5caca176e730);
hipFree(d_COUNT5caca176e730);
hipFree(d_BUF_5caca178f780);
hipFree(d_BUF_IDX_5caca178f780);
hipFree(d_COUNT5caca178f780);
hipFree(d_BUF_5caca1791330);
hipFree(d_BUF_IDX_5caca1791330);
hipFree(d_COUNT5caca1791330);
hipFree(d_KEY_5caca17441d0date__d_year);
hipFree(d_KEY_5caca17441d0part__p_brand1_encoded);
hipFree(d_KEY_5caca17441d0supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5caca1723900);
hipFree(d_MAT5caca1723900aggr0__tmp_attr0);
hipFree(d_MAT5caca1723900date__d_year);
hipFree(d_MAT5caca1723900part__p_brand1_encoded);
hipFree(d_MAT5caca1723900supplier__s_city_encoded);
hipFree(d_MAT_IDX5caca1723900);
free(MAT5caca1723900aggr0__tmp_attr0);
free(MAT5caca1723900date__d_year);
free(MAT5caca1723900part__p_brand1_encoded);
free(MAT5caca1723900supplier__s_city_encoded);
}