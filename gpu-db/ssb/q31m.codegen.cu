#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5eaa86934600(uint64_t* COUNT5eaa86950e20, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5eaa86950e20, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5eaa86934600(uint64_t* BUF_5eaa86950e20, uint64_t* BUF_IDX_5eaa86950e20, HASHTABLE_INSERT HT_5eaa86950e20, DBI32Type* customer__c_custkey, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5eaa86950e20 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5eaa86950e20 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_5eaa86950e20 = atomicAdd((int*)BUF_IDX_5eaa86950e20, 1);
HT_5eaa86950e20.insert(cuco::pair{KEY_5eaa86950e20, buf_idx_5eaa86950e20});
BUF_5eaa86950e20[buf_idx_5eaa86950e20 * 1 + 0] = tid;
}
__global__ void count_5eaa8695c8f0(uint64_t* COUNT5eaa869522e0, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5eaa869522e0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5eaa8695c8f0(uint64_t* BUF_5eaa869522e0, uint64_t* BUF_IDX_5eaa869522e0, HASHTABLE_INSERT HT_5eaa869522e0, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5eaa869522e0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5eaa869522e0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5eaa869522e0 = atomicAdd((int*)BUF_IDX_5eaa869522e0, 1);
HT_5eaa869522e0.insert(cuco::pair{KEY_5eaa869522e0, buf_idx_5eaa869522e0});
BUF_5eaa869522e0[buf_idx_5eaa869522e0 * 1 + 0] = tid;
}
__global__ void count_5eaa869626e0(uint64_t* COUNT5eaa8694fa30, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5eaa8694fa30, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5eaa869626e0(uint64_t* BUF_5eaa8694fa30, uint64_t* BUF_IDX_5eaa8694fa30, HASHTABLE_INSERT HT_5eaa8694fa30, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5eaa8694fa30 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5eaa8694fa30 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_5eaa8694fa30 = atomicAdd((int*)BUF_IDX_5eaa8694fa30, 1);
HT_5eaa8694fa30.insert(cuco::pair{KEY_5eaa8694fa30, buf_idx_5eaa8694fa30});
BUF_5eaa8694fa30[buf_idx_5eaa8694fa30 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5eaa86934ba0(uint64_t* BUF_5eaa8694fa30, uint64_t* BUF_5eaa86950e20, uint64_t* BUF_5eaa869522e0, HASHTABLE_INSERT HT_5eaa86904b70, HASHTABLE_PROBE HT_5eaa8694fa30, HASHTABLE_PROBE HT_5eaa86950e20, HASHTABLE_PROBE HT_5eaa869522e0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5eaa86950e20 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5eaa86950e20 |= reg_lineorder__lo_custkey;
//Probe Hash table
auto SLOT_5eaa86950e20 = HT_5eaa86950e20.find(KEY_5eaa86950e20);
if (SLOT_5eaa86950e20 == HT_5eaa86950e20.end()) return;
if (!(true)) return;
uint64_t KEY_5eaa869522e0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5eaa869522e0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_5eaa869522e0 = HT_5eaa869522e0.find(KEY_5eaa869522e0);
if (SLOT_5eaa869522e0 == HT_5eaa869522e0.end()) return;
if (!(true)) return;
uint64_t KEY_5eaa8694fa30 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5eaa8694fa30 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_5eaa8694fa30 = HT_5eaa8694fa30.find(KEY_5eaa8694fa30);
if (SLOT_5eaa8694fa30 == HT_5eaa8694fa30.end()) return;
if (!(true)) return;
uint64_t KEY_5eaa86904b70 = 0;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[BUF_5eaa86950e20[SLOT_5eaa86950e20->second * 1 + 0]];

KEY_5eaa86904b70 |= reg_customer__c_nation_encoded;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_5eaa869522e0[SLOT_5eaa869522e0->second * 1 + 0]];
KEY_5eaa86904b70 <<= 16;
KEY_5eaa86904b70 |= reg_supplier__s_nation_encoded;
auto reg_date__d_year = date__d_year[BUF_5eaa8694fa30[SLOT_5eaa8694fa30->second * 1 + 0]];
KEY_5eaa86904b70 <<= 32;
KEY_5eaa86904b70 |= reg_date__d_year;
//Create aggregation hash table
HT_5eaa86904b70.insert(cuco::pair{KEY_5eaa86904b70, 1});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5eaa86934ba0(uint64_t* BUF_5eaa8694fa30, uint64_t* BUF_5eaa86950e20, uint64_t* BUF_5eaa869522e0, HASHTABLE_FIND HT_5eaa86904b70, HASHTABLE_PROBE HT_5eaa8694fa30, HASHTABLE_PROBE HT_5eaa86950e20, HASHTABLE_PROBE HT_5eaa869522e0, DBI16Type* KEY_5eaa86904b70customer__c_nation_encoded, DBI32Type* KEY_5eaa86904b70date__d_year, DBI16Type* KEY_5eaa86904b70supplier__s_nation_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5eaa86950e20 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5eaa86950e20 |= reg_lineorder__lo_custkey;
//Probe Hash table
auto SLOT_5eaa86950e20 = HT_5eaa86950e20.find(KEY_5eaa86950e20);
if (SLOT_5eaa86950e20 == HT_5eaa86950e20.end()) return;
if (!(true)) return;
uint64_t KEY_5eaa869522e0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5eaa869522e0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_5eaa869522e0 = HT_5eaa869522e0.find(KEY_5eaa869522e0);
if (SLOT_5eaa869522e0 == HT_5eaa869522e0.end()) return;
if (!(true)) return;
uint64_t KEY_5eaa8694fa30 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5eaa8694fa30 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_5eaa8694fa30 = HT_5eaa8694fa30.find(KEY_5eaa8694fa30);
if (SLOT_5eaa8694fa30 == HT_5eaa8694fa30.end()) return;
if (!(true)) return;
uint64_t KEY_5eaa86904b70 = 0;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[BUF_5eaa86950e20[SLOT_5eaa86950e20->second * 1 + 0]];

KEY_5eaa86904b70 |= reg_customer__c_nation_encoded;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_5eaa869522e0[SLOT_5eaa869522e0->second * 1 + 0]];
KEY_5eaa86904b70 <<= 16;
KEY_5eaa86904b70 |= reg_supplier__s_nation_encoded;
auto reg_date__d_year = date__d_year[BUF_5eaa8694fa30[SLOT_5eaa8694fa30->second * 1 + 0]];
KEY_5eaa86904b70 <<= 32;
KEY_5eaa86904b70 |= reg_date__d_year;
//Aggregate in hashtable
auto buf_idx_5eaa86904b70 = HT_5eaa86904b70.find(KEY_5eaa86904b70)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5eaa86904b70], reg_lineorder__lo_revenue);
KEY_5eaa86904b70customer__c_nation_encoded[buf_idx_5eaa86904b70] = reg_customer__c_nation_encoded;
KEY_5eaa86904b70supplier__s_nation_encoded[buf_idx_5eaa86904b70] = reg_supplier__s_nation_encoded;
KEY_5eaa86904b70date__d_year[buf_idx_5eaa86904b70] = reg_date__d_year;
}
__global__ void count_5eaa8696f950(uint64_t* COUNT5eaa868e00c0, size_t COUNT5eaa86904b70) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5eaa86904b70) return;
//Materialize count
atomicAdd((int*)COUNT5eaa868e00c0, 1);
}
__global__ void main_5eaa8696f950(size_t COUNT5eaa86904b70, DBDecimalType* MAT5eaa868e00c0aggr0__tmp_attr0, DBI16Type* MAT5eaa868e00c0customer__c_nation_encoded, DBI32Type* MAT5eaa868e00c0date__d_year, DBI16Type* MAT5eaa868e00c0supplier__s_nation_encoded, uint64_t* MAT_IDX5eaa868e00c0, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5eaa86904b70) return;
//Materialize buffers
auto mat_idx5eaa868e00c0 = atomicAdd((int*)MAT_IDX5eaa868e00c0, 1);
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[tid];
MAT5eaa868e00c0customer__c_nation_encoded[mat_idx5eaa868e00c0] = reg_customer__c_nation_encoded;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[tid];
MAT5eaa868e00c0supplier__s_nation_encoded[mat_idx5eaa868e00c0] = reg_supplier__s_nation_encoded;
auto reg_date__d_year = date__d_year[tid];
MAT5eaa868e00c0date__d_year[mat_idx5eaa868e00c0] = reg_date__d_year;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5eaa868e00c0aggr0__tmp_attr0[mat_idx5eaa868e00c0] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT5eaa86950e20;
hipMalloc(&d_COUNT5eaa86950e20, sizeof(uint64_t));
hipMemset(d_COUNT5eaa86950e20, 0, sizeof(uint64_t));
count_5eaa86934600<<<std::ceil((float)customer_size/128.), 128>>>(d_COUNT5eaa86950e20, d_customer__c_region, customer_size);
uint64_t COUNT5eaa86950e20;
hipMemcpy(&COUNT5eaa86950e20, d_COUNT5eaa86950e20, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5eaa86950e20;
hipMalloc(&d_BUF_IDX_5eaa86950e20, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5eaa86950e20, 0, sizeof(uint64_t));
uint64_t* d_BUF_5eaa86950e20;
hipMalloc(&d_BUF_5eaa86950e20, sizeof(uint64_t) * COUNT5eaa86950e20 * 1);
auto d_HT_5eaa86950e20 = cuco::static_map{ (int)COUNT5eaa86950e20*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5eaa86934600<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_5eaa86950e20, d_BUF_IDX_5eaa86950e20, d_HT_5eaa86950e20.ref(cuco::insert), d_customer__c_custkey, d_customer__c_region, customer_size);
//Materialize count
uint64_t* d_COUNT5eaa869522e0;
hipMalloc(&d_COUNT5eaa869522e0, sizeof(uint64_t));
hipMemset(d_COUNT5eaa869522e0, 0, sizeof(uint64_t));
count_5eaa8695c8f0<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT5eaa869522e0, d_supplier__s_region, supplier_size);
uint64_t COUNT5eaa869522e0;
hipMemcpy(&COUNT5eaa869522e0, d_COUNT5eaa869522e0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5eaa869522e0;
hipMalloc(&d_BUF_IDX_5eaa869522e0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5eaa869522e0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5eaa869522e0;
hipMalloc(&d_BUF_5eaa869522e0, sizeof(uint64_t) * COUNT5eaa869522e0 * 1);
auto d_HT_5eaa869522e0 = cuco::static_map{ (int)COUNT5eaa869522e0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5eaa8695c8f0<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_5eaa869522e0, d_BUF_IDX_5eaa869522e0, d_HT_5eaa869522e0.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5eaa8694fa30;
hipMalloc(&d_COUNT5eaa8694fa30, sizeof(uint64_t));
hipMemset(d_COUNT5eaa8694fa30, 0, sizeof(uint64_t));
count_5eaa869626e0<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT5eaa8694fa30, d_date__d_year, date_size);
uint64_t COUNT5eaa8694fa30;
hipMemcpy(&COUNT5eaa8694fa30, d_COUNT5eaa8694fa30, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5eaa8694fa30;
hipMalloc(&d_BUF_IDX_5eaa8694fa30, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5eaa8694fa30, 0, sizeof(uint64_t));
uint64_t* d_BUF_5eaa8694fa30;
hipMalloc(&d_BUF_5eaa8694fa30, sizeof(uint64_t) * COUNT5eaa8694fa30 * 1);
auto d_HT_5eaa8694fa30 = cuco::static_map{ (int)COUNT5eaa8694fa30*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5eaa869626e0<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_5eaa8694fa30, d_BUF_IDX_5eaa8694fa30, d_HT_5eaa8694fa30.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Create aggregation hash table
auto d_HT_5eaa86904b70 = cuco::static_map{ (int)144285*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5eaa86934ba0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5eaa8694fa30, d_BUF_5eaa86950e20, d_BUF_5eaa869522e0, d_HT_5eaa86904b70.ref(cuco::insert), d_HT_5eaa8694fa30.ref(cuco::find), d_HT_5eaa86950e20.ref(cuco::find), d_HT_5eaa869522e0.ref(cuco::find), d_customer__c_nation_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_nation_encoded);
size_t COUNT5eaa86904b70 = d_HT_5eaa86904b70.size();
thrust::device_vector<int64_t> keys_5eaa86904b70(COUNT5eaa86904b70), vals_5eaa86904b70(COUNT5eaa86904b70);
d_HT_5eaa86904b70.retrieve_all(keys_5eaa86904b70.begin(), vals_5eaa86904b70.begin());
d_HT_5eaa86904b70.clear();
int64_t* raw_keys5eaa86904b70 = thrust::raw_pointer_cast(keys_5eaa86904b70.data());
insertKeys<<<std::ceil((float)COUNT5eaa86904b70/128.), 128>>>(raw_keys5eaa86904b70, d_HT_5eaa86904b70.ref(cuco::insert), COUNT5eaa86904b70);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5eaa86904b70);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5eaa86904b70);
DBI16Type* d_KEY_5eaa86904b70customer__c_nation_encoded;
hipMalloc(&d_KEY_5eaa86904b70customer__c_nation_encoded, sizeof(DBI16Type) * COUNT5eaa86904b70);
hipMemset(d_KEY_5eaa86904b70customer__c_nation_encoded, 0, sizeof(DBI16Type) * COUNT5eaa86904b70);
DBI16Type* d_KEY_5eaa86904b70supplier__s_nation_encoded;
hipMalloc(&d_KEY_5eaa86904b70supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT5eaa86904b70);
hipMemset(d_KEY_5eaa86904b70supplier__s_nation_encoded, 0, sizeof(DBI16Type) * COUNT5eaa86904b70);
DBI32Type* d_KEY_5eaa86904b70date__d_year;
hipMalloc(&d_KEY_5eaa86904b70date__d_year, sizeof(DBI32Type) * COUNT5eaa86904b70);
hipMemset(d_KEY_5eaa86904b70date__d_year, 0, sizeof(DBI32Type) * COUNT5eaa86904b70);
main_5eaa86934ba0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5eaa8694fa30, d_BUF_5eaa86950e20, d_BUF_5eaa869522e0, d_HT_5eaa86904b70.ref(cuco::find), d_HT_5eaa8694fa30.ref(cuco::find), d_HT_5eaa86950e20.ref(cuco::find), d_HT_5eaa869522e0.ref(cuco::find), d_KEY_5eaa86904b70customer__c_nation_encoded, d_KEY_5eaa86904b70date__d_year, d_KEY_5eaa86904b70supplier__s_nation_encoded, d_aggr0__tmp_attr0, d_customer__c_nation_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_nation_encoded);
//Materialize count
uint64_t* d_COUNT5eaa868e00c0;
hipMalloc(&d_COUNT5eaa868e00c0, sizeof(uint64_t));
hipMemset(d_COUNT5eaa868e00c0, 0, sizeof(uint64_t));
count_5eaa8696f950<<<std::ceil((float)COUNT5eaa86904b70/128.), 128>>>(d_COUNT5eaa868e00c0, COUNT5eaa86904b70);
uint64_t COUNT5eaa868e00c0;
hipMemcpy(&COUNT5eaa868e00c0, d_COUNT5eaa868e00c0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5eaa868e00c0;
hipMalloc(&d_MAT_IDX5eaa868e00c0, sizeof(uint64_t));
hipMemset(d_MAT_IDX5eaa868e00c0, 0, sizeof(uint64_t));
auto MAT5eaa868e00c0customer__c_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5eaa868e00c0);
DBI16Type* d_MAT5eaa868e00c0customer__c_nation_encoded;
hipMalloc(&d_MAT5eaa868e00c0customer__c_nation_encoded, sizeof(DBI16Type) * COUNT5eaa868e00c0);
auto MAT5eaa868e00c0supplier__s_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5eaa868e00c0);
DBI16Type* d_MAT5eaa868e00c0supplier__s_nation_encoded;
hipMalloc(&d_MAT5eaa868e00c0supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT5eaa868e00c0);
auto MAT5eaa868e00c0date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5eaa868e00c0);
DBI32Type* d_MAT5eaa868e00c0date__d_year;
hipMalloc(&d_MAT5eaa868e00c0date__d_year, sizeof(DBI32Type) * COUNT5eaa868e00c0);
auto MAT5eaa868e00c0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5eaa868e00c0);
DBDecimalType* d_MAT5eaa868e00c0aggr0__tmp_attr0;
hipMalloc(&d_MAT5eaa868e00c0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5eaa868e00c0);
main_5eaa8696f950<<<std::ceil((float)COUNT5eaa86904b70/128.), 128>>>(COUNT5eaa86904b70, d_MAT5eaa868e00c0aggr0__tmp_attr0, d_MAT5eaa868e00c0customer__c_nation_encoded, d_MAT5eaa868e00c0date__d_year, d_MAT5eaa868e00c0supplier__s_nation_encoded, d_MAT_IDX5eaa868e00c0, d_aggr0__tmp_attr0, d_KEY_5eaa86904b70customer__c_nation_encoded, d_KEY_5eaa86904b70date__d_year, d_KEY_5eaa86904b70supplier__s_nation_encoded);
hipMemcpy(MAT5eaa868e00c0customer__c_nation_encoded, d_MAT5eaa868e00c0customer__c_nation_encoded, sizeof(DBI16Type) * COUNT5eaa868e00c0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5eaa868e00c0supplier__s_nation_encoded, d_MAT5eaa868e00c0supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT5eaa868e00c0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5eaa868e00c0date__d_year, d_MAT5eaa868e00c0date__d_year, sizeof(DBI32Type) * COUNT5eaa868e00c0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5eaa868e00c0aggr0__tmp_attr0, d_MAT5eaa868e00c0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5eaa868e00c0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5eaa868e00c0; i++) { std::cout << "" << customer__c_nation_map[MAT5eaa868e00c0customer__c_nation_encoded[i]];
std::cout << "," << supplier__s_nation_map[MAT5eaa868e00c0supplier__s_nation_encoded[i]];
std::cout << "," << MAT5eaa868e00c0date__d_year[i];
std::cout << "," << MAT5eaa868e00c0aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_5eaa86950e20);
hipFree(d_BUF_IDX_5eaa86950e20);
hipFree(d_COUNT5eaa86950e20);
hipFree(d_BUF_5eaa869522e0);
hipFree(d_BUF_IDX_5eaa869522e0);
hipFree(d_COUNT5eaa869522e0);
hipFree(d_BUF_5eaa8694fa30);
hipFree(d_BUF_IDX_5eaa8694fa30);
hipFree(d_COUNT5eaa8694fa30);
hipFree(d_KEY_5eaa86904b70customer__c_nation_encoded);
hipFree(d_KEY_5eaa86904b70date__d_year);
hipFree(d_KEY_5eaa86904b70supplier__s_nation_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5eaa868e00c0);
hipFree(d_MAT5eaa868e00c0aggr0__tmp_attr0);
hipFree(d_MAT5eaa868e00c0customer__c_nation_encoded);
hipFree(d_MAT5eaa868e00c0date__d_year);
hipFree(d_MAT5eaa868e00c0supplier__s_nation_encoded);
hipFree(d_MAT_IDX5eaa868e00c0);
free(MAT5eaa868e00c0aggr0__tmp_attr0);
free(MAT5eaa868e00c0customer__c_nation_encoded);
free(MAT5eaa868e00c0date__d_year);
free(MAT5eaa868e00c0supplier__s_nation_encoded);
}