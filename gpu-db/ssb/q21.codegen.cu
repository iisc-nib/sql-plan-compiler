#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5cdcaba87b60(uint64_t* COUNT5cdcabaa5db0, DBStringType* part__p_category, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_category = part__p_category[tid];
if (!(evaluatePredicate(reg_part__p_category, "MFGR#12", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5cdcabaa5db0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5cdcaba87b60(uint64_t* BUF_5cdcabaa5db0, uint64_t* BUF_IDX_5cdcabaa5db0, HASHTABLE_INSERT HT_5cdcabaa5db0, DBStringType* part__p_category, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_category = part__p_category[tid];
if (!(evaluatePredicate(reg_part__p_category, "MFGR#12", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5cdcabaa5db0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_5cdcabaa5db0 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_5cdcabaa5db0 = atomicAdd((int*)BUF_IDX_5cdcabaa5db0, 1);
HT_5cdcabaa5db0.insert(cuco::pair{KEY_5cdcabaa5db0, buf_idx_5cdcabaa5db0});
BUF_5cdcabaa5db0[buf_idx_5cdcabaa5db0 * 1 + 0] = tid;
}
__global__ void count_5cdcabab15c0(uint64_t* COUNT5cdcabaa0080, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5cdcabaa0080, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5cdcabab15c0(uint64_t* BUF_5cdcabaa0080, uint64_t* BUF_IDX_5cdcabaa0080, HASHTABLE_INSERT HT_5cdcabaa0080, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5cdcabaa0080 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5cdcabaa0080 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5cdcabaa0080 = atomicAdd((int*)BUF_IDX_5cdcabaa0080, 1);
HT_5cdcabaa0080.insert(cuco::pair{KEY_5cdcabaa0080, buf_idx_5cdcabaa0080});
BUF_5cdcabaa0080[buf_idx_5cdcabaa0080 * 1 + 0] = tid;
}
__global__ void count_5cdcaba87590(uint64_t* COUNT5cdcabaa3680, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5cdcabaa3680, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5cdcaba87590(uint64_t* BUF_5cdcabaa3680, uint64_t* BUF_IDX_5cdcabaa3680, HASHTABLE_INSERT HT_5cdcabaa3680, DBI32Type* date__d_datekey, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5cdcabaa3680 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5cdcabaa3680 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_5cdcabaa3680 = atomicAdd((int*)BUF_IDX_5cdcabaa3680, 1);
HT_5cdcabaa3680.insert(cuco::pair{KEY_5cdcabaa3680, buf_idx_5cdcabaa3680});
BUF_5cdcabaa3680[buf_idx_5cdcabaa3680 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5cdcaba9dfe0(uint64_t* BUF_5cdcabaa0080, uint64_t* BUF_5cdcabaa3680, uint64_t* BUF_5cdcabaa5db0, HASHTABLE_INSERT HT_5cdcaba58400, HASHTABLE_PROBE HT_5cdcabaa0080, HASHTABLE_PROBE HT_5cdcabaa3680, HASHTABLE_PROBE HT_5cdcabaa5db0, DBI32Type* date__d_year, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5cdcabaa5db0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5cdcabaa5db0 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5cdcabaa5db0.for_each(KEY_5cdcabaa5db0, [&] __device__ (auto const SLOT_5cdcabaa5db0) {

auto const [slot_first5cdcabaa5db0, slot_second5cdcabaa5db0] = SLOT_5cdcabaa5db0;
if (!(true)) return;
uint64_t KEY_5cdcabaa0080 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5cdcabaa0080 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5cdcabaa0080.for_each(KEY_5cdcabaa0080, [&] __device__ (auto const SLOT_5cdcabaa0080) {

auto const [slot_first5cdcabaa0080, slot_second5cdcabaa0080] = SLOT_5cdcabaa0080;
if (!(true)) return;
uint64_t KEY_5cdcabaa3680 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5cdcabaa3680 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5cdcabaa3680.for_each(KEY_5cdcabaa3680, [&] __device__ (auto const SLOT_5cdcabaa3680) {

auto const [slot_first5cdcabaa3680, slot_second5cdcabaa3680] = SLOT_5cdcabaa3680;
if (!(true)) return;
uint64_t KEY_5cdcaba58400 = 0;
auto reg_date__d_year = date__d_year[BUF_5cdcabaa3680[slot_second5cdcabaa3680 * 1 + 0]];

KEY_5cdcaba58400 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_5cdcabaa5db0[slot_second5cdcabaa5db0 * 1 + 0]];
KEY_5cdcaba58400 <<= 16;
KEY_5cdcaba58400 |= reg_part__p_brand1_encoded;
//Create aggregation hash table
HT_5cdcaba58400.insert(cuco::pair{KEY_5cdcaba58400, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5cdcaba9dfe0(uint64_t* BUF_5cdcabaa0080, uint64_t* BUF_5cdcabaa3680, uint64_t* BUF_5cdcabaa5db0, HASHTABLE_FIND HT_5cdcaba58400, HASHTABLE_PROBE HT_5cdcabaa0080, HASHTABLE_PROBE HT_5cdcabaa3680, HASHTABLE_PROBE HT_5cdcabaa5db0, DBI32Type* KEY_5cdcaba58400date__d_year, DBI16Type* KEY_5cdcaba58400part__p_brand1_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5cdcabaa5db0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5cdcabaa5db0 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5cdcabaa5db0.for_each(KEY_5cdcabaa5db0, [&] __device__ (auto const SLOT_5cdcabaa5db0) {
auto const [slot_first5cdcabaa5db0, slot_second5cdcabaa5db0] = SLOT_5cdcabaa5db0;
if (!(true)) return;
uint64_t KEY_5cdcabaa0080 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5cdcabaa0080 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5cdcabaa0080.for_each(KEY_5cdcabaa0080, [&] __device__ (auto const SLOT_5cdcabaa0080) {
auto const [slot_first5cdcabaa0080, slot_second5cdcabaa0080] = SLOT_5cdcabaa0080;
if (!(true)) return;
uint64_t KEY_5cdcabaa3680 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5cdcabaa3680 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5cdcabaa3680.for_each(KEY_5cdcabaa3680, [&] __device__ (auto const SLOT_5cdcabaa3680) {
auto const [slot_first5cdcabaa3680, slot_second5cdcabaa3680] = SLOT_5cdcabaa3680;
if (!(true)) return;
uint64_t KEY_5cdcaba58400 = 0;
auto reg_date__d_year = date__d_year[BUF_5cdcabaa3680[slot_second5cdcabaa3680 * 1 + 0]];

KEY_5cdcaba58400 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_5cdcabaa5db0[slot_second5cdcabaa5db0 * 1 + 0]];
KEY_5cdcaba58400 <<= 16;
KEY_5cdcaba58400 |= reg_part__p_brand1_encoded;
//Aggregate in hashtable
auto buf_idx_5cdcaba58400 = HT_5cdcaba58400.find(KEY_5cdcaba58400)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5cdcaba58400], reg_lineorder__lo_revenue);
KEY_5cdcaba58400date__d_year[buf_idx_5cdcaba58400] = reg_date__d_year;
KEY_5cdcaba58400part__p_brand1_encoded[buf_idx_5cdcaba58400] = reg_part__p_brand1_encoded;
});
});
});
}
__global__ void count_5cdcabac08a0(size_t COUNT5cdcaba58400, uint64_t* COUNT5cdcaba6b580) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5cdcaba58400) return;
//Materialize count
atomicAdd((int*)COUNT5cdcaba6b580, 1);
}
__global__ void main_5cdcabac08a0(size_t COUNT5cdcaba58400, DBDecimalType* MAT5cdcaba6b580aggr0__tmp_attr0, DBI32Type* MAT5cdcaba6b580date__d_year, DBI16Type* MAT5cdcaba6b580part__p_brand1_encoded, uint64_t* MAT_IDX5cdcaba6b580, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5cdcaba58400) return;
//Materialize buffers
auto mat_idx5cdcaba6b580 = atomicAdd((int*)MAT_IDX5cdcaba6b580, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5cdcaba6b580aggr0__tmp_attr0[mat_idx5cdcaba6b580] = reg_aggr0__tmp_attr0;
auto reg_date__d_year = date__d_year[tid];
MAT5cdcaba6b580date__d_year[mat_idx5cdcaba6b580] = reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[tid];
MAT5cdcaba6b580part__p_brand1_encoded[mat_idx5cdcaba6b580] = reg_part__p_brand1_encoded;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT5cdcabaa5db0;
hipMalloc(&d_COUNT5cdcabaa5db0, sizeof(uint64_t));
hipMemset(d_COUNT5cdcabaa5db0, 0, sizeof(uint64_t));
count_5cdcaba87b60<<<std::ceil((float)part_size/128.), 128>>>(d_COUNT5cdcabaa5db0, d_part__p_category, part_size);
uint64_t COUNT5cdcabaa5db0;
hipMemcpy(&COUNT5cdcabaa5db0, d_COUNT5cdcabaa5db0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5cdcabaa5db0;
hipMalloc(&d_BUF_IDX_5cdcabaa5db0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5cdcabaa5db0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5cdcabaa5db0;
hipMalloc(&d_BUF_5cdcabaa5db0, sizeof(uint64_t) * COUNT5cdcabaa5db0 * 1);
auto d_HT_5cdcabaa5db0 = cuco::experimental::static_multimap{ (int)COUNT5cdcabaa5db0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5cdcaba87b60<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_5cdcabaa5db0, d_BUF_IDX_5cdcabaa5db0, d_HT_5cdcabaa5db0.ref(cuco::insert), d_part__p_category, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT5cdcabaa0080;
hipMalloc(&d_COUNT5cdcabaa0080, sizeof(uint64_t));
hipMemset(d_COUNT5cdcabaa0080, 0, sizeof(uint64_t));
count_5cdcabab15c0<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT5cdcabaa0080, d_supplier__s_region, supplier_size);
uint64_t COUNT5cdcabaa0080;
hipMemcpy(&COUNT5cdcabaa0080, d_COUNT5cdcabaa0080, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5cdcabaa0080;
hipMalloc(&d_BUF_IDX_5cdcabaa0080, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5cdcabaa0080, 0, sizeof(uint64_t));
uint64_t* d_BUF_5cdcabaa0080;
hipMalloc(&d_BUF_5cdcabaa0080, sizeof(uint64_t) * COUNT5cdcabaa0080 * 1);
auto d_HT_5cdcabaa0080 = cuco::experimental::static_multimap{ (int)COUNT5cdcabaa0080*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5cdcabab15c0<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_5cdcabaa0080, d_BUF_IDX_5cdcabaa0080, d_HT_5cdcabaa0080.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5cdcabaa3680;
hipMalloc(&d_COUNT5cdcabaa3680, sizeof(uint64_t));
hipMemset(d_COUNT5cdcabaa3680, 0, sizeof(uint64_t));
count_5cdcaba87590<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT5cdcabaa3680, date_size);
uint64_t COUNT5cdcabaa3680;
hipMemcpy(&COUNT5cdcabaa3680, d_COUNT5cdcabaa3680, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5cdcabaa3680;
hipMalloc(&d_BUF_IDX_5cdcabaa3680, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5cdcabaa3680, 0, sizeof(uint64_t));
uint64_t* d_BUF_5cdcabaa3680;
hipMalloc(&d_BUF_5cdcabaa3680, sizeof(uint64_t) * COUNT5cdcabaa3680 * 1);
auto d_HT_5cdcabaa3680 = cuco::experimental::static_multimap{ (int)COUNT5cdcabaa3680*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5cdcaba87590<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_5cdcabaa3680, d_BUF_IDX_5cdcabaa3680, d_HT_5cdcabaa3680.ref(cuco::insert), d_date__d_datekey, date_size);
//Create aggregation hash table
auto d_HT_5cdcaba58400 = cuco::static_map{ (int)52974*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5cdcaba9dfe0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5cdcabaa0080, d_BUF_5cdcabaa3680, d_BUF_5cdcabaa5db0, d_HT_5cdcaba58400.ref(cuco::insert), d_HT_5cdcabaa0080.ref(cuco::for_each), d_HT_5cdcabaa3680.ref(cuco::for_each), d_HT_5cdcabaa5db0.ref(cuco::for_each), d_date__d_year, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size, d_part__p_brand1_encoded);
size_t COUNT5cdcaba58400 = d_HT_5cdcaba58400.size();
thrust::device_vector<int64_t> keys_5cdcaba58400(COUNT5cdcaba58400), vals_5cdcaba58400(COUNT5cdcaba58400);
d_HT_5cdcaba58400.retrieve_all(keys_5cdcaba58400.begin(), vals_5cdcaba58400.begin());
d_HT_5cdcaba58400.clear();
int64_t* raw_keys5cdcaba58400 = thrust::raw_pointer_cast(keys_5cdcaba58400.data());
insertKeys<<<std::ceil((float)COUNT5cdcaba58400/128.), 128>>>(raw_keys5cdcaba58400, d_HT_5cdcaba58400.ref(cuco::insert), COUNT5cdcaba58400);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5cdcaba58400);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5cdcaba58400);
DBI32Type* d_KEY_5cdcaba58400date__d_year;
hipMalloc(&d_KEY_5cdcaba58400date__d_year, sizeof(DBI32Type) * COUNT5cdcaba58400);
hipMemset(d_KEY_5cdcaba58400date__d_year, 0, sizeof(DBI32Type) * COUNT5cdcaba58400);
DBI16Type* d_KEY_5cdcaba58400part__p_brand1_encoded;
hipMalloc(&d_KEY_5cdcaba58400part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5cdcaba58400);
hipMemset(d_KEY_5cdcaba58400part__p_brand1_encoded, 0, sizeof(DBI16Type) * COUNT5cdcaba58400);
main_5cdcaba9dfe0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5cdcabaa0080, d_BUF_5cdcabaa3680, d_BUF_5cdcabaa5db0, d_HT_5cdcaba58400.ref(cuco::find), d_HT_5cdcabaa0080.ref(cuco::for_each), d_HT_5cdcabaa3680.ref(cuco::for_each), d_HT_5cdcabaa5db0.ref(cuco::for_each), d_KEY_5cdcaba58400date__d_year, d_KEY_5cdcaba58400part__p_brand1_encoded, d_aggr0__tmp_attr0, d_date__d_year, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_part__p_brand1_encoded);
//Materialize count
uint64_t* d_COUNT5cdcaba6b580;
hipMalloc(&d_COUNT5cdcaba6b580, sizeof(uint64_t));
hipMemset(d_COUNT5cdcaba6b580, 0, sizeof(uint64_t));
count_5cdcabac08a0<<<std::ceil((float)COUNT5cdcaba58400/128.), 128>>>(COUNT5cdcaba58400, d_COUNT5cdcaba6b580);
uint64_t COUNT5cdcaba6b580;
hipMemcpy(&COUNT5cdcaba6b580, d_COUNT5cdcaba6b580, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5cdcaba6b580;
hipMalloc(&d_MAT_IDX5cdcaba6b580, sizeof(uint64_t));
hipMemset(d_MAT_IDX5cdcaba6b580, 0, sizeof(uint64_t));
auto MAT5cdcaba6b580aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5cdcaba6b580);
DBDecimalType* d_MAT5cdcaba6b580aggr0__tmp_attr0;
hipMalloc(&d_MAT5cdcaba6b580aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5cdcaba6b580);
auto MAT5cdcaba6b580date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5cdcaba6b580);
DBI32Type* d_MAT5cdcaba6b580date__d_year;
hipMalloc(&d_MAT5cdcaba6b580date__d_year, sizeof(DBI32Type) * COUNT5cdcaba6b580);
auto MAT5cdcaba6b580part__p_brand1_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5cdcaba6b580);
DBI16Type* d_MAT5cdcaba6b580part__p_brand1_encoded;
hipMalloc(&d_MAT5cdcaba6b580part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5cdcaba6b580);
main_5cdcabac08a0<<<std::ceil((float)COUNT5cdcaba58400/128.), 128>>>(COUNT5cdcaba58400, d_MAT5cdcaba6b580aggr0__tmp_attr0, d_MAT5cdcaba6b580date__d_year, d_MAT5cdcaba6b580part__p_brand1_encoded, d_MAT_IDX5cdcaba6b580, d_aggr0__tmp_attr0, d_KEY_5cdcaba58400date__d_year, d_KEY_5cdcaba58400part__p_brand1_encoded);
hipMemcpy(MAT5cdcaba6b580aggr0__tmp_attr0, d_MAT5cdcaba6b580aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5cdcaba6b580, hipMemcpyDeviceToHost);
hipMemcpy(MAT5cdcaba6b580date__d_year, d_MAT5cdcaba6b580date__d_year, sizeof(DBI32Type) * COUNT5cdcaba6b580, hipMemcpyDeviceToHost);
hipMemcpy(MAT5cdcaba6b580part__p_brand1_encoded, d_MAT5cdcaba6b580part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5cdcaba6b580, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5cdcaba6b580; i++) { std::cout << "" << MAT5cdcaba6b580aggr0__tmp_attr0[i];
std::cout << "," << MAT5cdcaba6b580date__d_year[i];
std::cout << "," << part__p_brand1_map[MAT5cdcaba6b580part__p_brand1_encoded[i]];
std::cout << std::endl; }
hipFree(d_BUF_5cdcabaa5db0);
hipFree(d_BUF_IDX_5cdcabaa5db0);
hipFree(d_COUNT5cdcabaa5db0);
hipFree(d_BUF_5cdcabaa0080);
hipFree(d_BUF_IDX_5cdcabaa0080);
hipFree(d_COUNT5cdcabaa0080);
hipFree(d_BUF_5cdcabaa3680);
hipFree(d_BUF_IDX_5cdcabaa3680);
hipFree(d_COUNT5cdcabaa3680);
hipFree(d_KEY_5cdcaba58400date__d_year);
hipFree(d_KEY_5cdcaba58400part__p_brand1_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5cdcaba6b580);
hipFree(d_MAT5cdcaba6b580aggr0__tmp_attr0);
hipFree(d_MAT5cdcaba6b580date__d_year);
hipFree(d_MAT5cdcaba6b580part__p_brand1_encoded);
hipFree(d_MAT_IDX5cdcaba6b580);
free(MAT5cdcaba6b580aggr0__tmp_attr0);
free(MAT5cdcaba6b580date__d_year);
free(MAT5cdcaba6b580part__p_brand1_encoded);
}