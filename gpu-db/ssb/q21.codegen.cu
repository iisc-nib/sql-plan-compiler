#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_57fc10a61860(uint64_t* COUNT57fc10a56d10, DBStringType* part__p_category, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_category = part__p_category[tid];
if (!(evaluatePredicate(reg_part__p_category, "MFGR#12", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT57fc10a56d10, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_57fc10a61860(uint64_t* BUF_57fc10a56d10, uint64_t* BUF_IDX_57fc10a56d10, HASHTABLE_INSERT HT_57fc10a56d10, DBStringType* part__p_category, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_category = part__p_category[tid];
if (!(evaluatePredicate(reg_part__p_category, "MFGR#12", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_57fc10a56d10 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_57fc10a56d10 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_57fc10a56d10 = atomicAdd((int*)BUF_IDX_57fc10a56d10, 1);
HT_57fc10a56d10.insert(cuco::pair{KEY_57fc10a56d10, buf_idx_57fc10a56d10});
BUF_57fc10a56d10[buf_idx_57fc10a56d10 * 1 + 0] = tid;
}
__global__ void count_57fc10a64300(uint64_t* COUNT57fc10a51c00, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT57fc10a51c00, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_57fc10a64300(uint64_t* BUF_57fc10a51c00, uint64_t* BUF_IDX_57fc10a51c00, HASHTABLE_INSERT HT_57fc10a51c00, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_57fc10a51c00 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_57fc10a51c00 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_57fc10a51c00 = atomicAdd((int*)BUF_IDX_57fc10a51c00, 1);
HT_57fc10a51c00.insert(cuco::pair{KEY_57fc10a51c00, buf_idx_57fc10a51c00});
BUF_57fc10a51c00[buf_idx_57fc10a51c00 * 1 + 0] = tid;
}
__global__ void count_57fc10a3a5e0(uint64_t* COUNT57fc10a59000, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT57fc10a59000, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_57fc10a3a5e0(uint64_t* BUF_57fc10a59000, uint64_t* BUF_IDX_57fc10a59000, HASHTABLE_INSERT HT_57fc10a59000, DBI32Type* date__d_datekey, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_57fc10a59000 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_57fc10a59000 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_57fc10a59000 = atomicAdd((int*)BUF_IDX_57fc10a59000, 1);
HT_57fc10a59000.insert(cuco::pair{KEY_57fc10a59000, buf_idx_57fc10a59000});
BUF_57fc10a59000[buf_idx_57fc10a59000 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_57fc10a3a010(uint64_t* BUF_57fc10a51c00, uint64_t* BUF_57fc10a56d10, uint64_t* BUF_57fc10a59000, HASHTABLE_INSERT HT_57fc10a0b2c0, HASHTABLE_PROBE HT_57fc10a51c00, HASHTABLE_PROBE HT_57fc10a56d10, HASHTABLE_PROBE HT_57fc10a59000, DBI32Type* date__d_year, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_57fc10a56d10 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_57fc10a56d10 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_57fc10a56d10.for_each(KEY_57fc10a56d10, [&] __device__ (auto const SLOT_57fc10a56d10) {

auto const [slot_first57fc10a56d10, slot_second57fc10a56d10] = SLOT_57fc10a56d10;
if (!(true)) return;
uint64_t KEY_57fc10a51c00 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_57fc10a51c00 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_57fc10a51c00.for_each(KEY_57fc10a51c00, [&] __device__ (auto const SLOT_57fc10a51c00) {

auto const [slot_first57fc10a51c00, slot_second57fc10a51c00] = SLOT_57fc10a51c00;
if (!(true)) return;
uint64_t KEY_57fc10a59000 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_57fc10a59000 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_57fc10a59000.for_each(KEY_57fc10a59000, [&] __device__ (auto const SLOT_57fc10a59000) {

auto const [slot_first57fc10a59000, slot_second57fc10a59000] = SLOT_57fc10a59000;
if (!(true)) return;
uint64_t KEY_57fc10a0b2c0 = 0;
auto reg_date__d_year = date__d_year[BUF_57fc10a59000[slot_second57fc10a59000 * 1 + 0]];

KEY_57fc10a0b2c0 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_57fc10a56d10[slot_second57fc10a56d10 * 1 + 0]];
KEY_57fc10a0b2c0 <<= 16;
KEY_57fc10a0b2c0 |= reg_part__p_brand1_encoded;
//Create aggregation hash table
HT_57fc10a0b2c0.insert(cuco::pair{KEY_57fc10a0b2c0, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_57fc10a3a010(uint64_t* BUF_57fc10a51c00, uint64_t* BUF_57fc10a56d10, uint64_t* BUF_57fc10a59000, HASHTABLE_FIND HT_57fc10a0b2c0, HASHTABLE_PROBE HT_57fc10a51c00, HASHTABLE_PROBE HT_57fc10a56d10, HASHTABLE_PROBE HT_57fc10a59000, DBI32Type* KEY_57fc10a0b2c0date__d_year, DBI16Type* KEY_57fc10a0b2c0part__p_brand1_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_57fc10a56d10 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_57fc10a56d10 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_57fc10a56d10.for_each(KEY_57fc10a56d10, [&] __device__ (auto const SLOT_57fc10a56d10) {
auto const [slot_first57fc10a56d10, slot_second57fc10a56d10] = SLOT_57fc10a56d10;
if (!(true)) return;
uint64_t KEY_57fc10a51c00 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_57fc10a51c00 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_57fc10a51c00.for_each(KEY_57fc10a51c00, [&] __device__ (auto const SLOT_57fc10a51c00) {
auto const [slot_first57fc10a51c00, slot_second57fc10a51c00] = SLOT_57fc10a51c00;
if (!(true)) return;
uint64_t KEY_57fc10a59000 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_57fc10a59000 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_57fc10a59000.for_each(KEY_57fc10a59000, [&] __device__ (auto const SLOT_57fc10a59000) {
auto const [slot_first57fc10a59000, slot_second57fc10a59000] = SLOT_57fc10a59000;
if (!(true)) return;
uint64_t KEY_57fc10a0b2c0 = 0;
auto reg_date__d_year = date__d_year[BUF_57fc10a59000[slot_second57fc10a59000 * 1 + 0]];

KEY_57fc10a0b2c0 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_57fc10a56d10[slot_second57fc10a56d10 * 1 + 0]];
KEY_57fc10a0b2c0 <<= 16;
KEY_57fc10a0b2c0 |= reg_part__p_brand1_encoded;
//Aggregate in hashtable
auto buf_idx_57fc10a0b2c0 = HT_57fc10a0b2c0.find(KEY_57fc10a0b2c0)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_57fc10a0b2c0], reg_lineorder__lo_revenue);
KEY_57fc10a0b2c0date__d_year[buf_idx_57fc10a0b2c0] = reg_date__d_year;
KEY_57fc10a0b2c0part__p_brand1_encoded[buf_idx_57fc10a0b2c0] = reg_part__p_brand1_encoded;
});
});
});
}
__global__ void count_57fc10a73600(size_t COUNT57fc10a0b2c0, uint64_t* COUNT57fc10a1e470) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT57fc10a0b2c0) return;
//Materialize count
atomicAdd((int*)COUNT57fc10a1e470, 1);
}
__global__ void main_57fc10a73600(size_t COUNT57fc10a0b2c0, DBDecimalType* MAT57fc10a1e470aggr0__tmp_attr0, DBI32Type* MAT57fc10a1e470date__d_year, DBI16Type* MAT57fc10a1e470part__p_brand1_encoded, uint64_t* MAT_IDX57fc10a1e470, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT57fc10a0b2c0) return;
//Materialize buffers
auto mat_idx57fc10a1e470 = atomicAdd((int*)MAT_IDX57fc10a1e470, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT57fc10a1e470aggr0__tmp_attr0[mat_idx57fc10a1e470] = reg_aggr0__tmp_attr0;
auto reg_date__d_year = date__d_year[tid];
MAT57fc10a1e470date__d_year[mat_idx57fc10a1e470] = reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[tid];
MAT57fc10a1e470part__p_brand1_encoded[mat_idx57fc10a1e470] = reg_part__p_brand1_encoded;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT57fc10a56d10;
hipMalloc(&d_COUNT57fc10a56d10, sizeof(uint64_t));
hipMemset(d_COUNT57fc10a56d10, 0, sizeof(uint64_t));
count_57fc10a61860<<<std::ceil((float)part_size/32.), 32>>>(d_COUNT57fc10a56d10, d_part__p_category, part_size);
uint64_t COUNT57fc10a56d10;
hipMemcpy(&COUNT57fc10a56d10, d_COUNT57fc10a56d10, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_57fc10a56d10;
hipMalloc(&d_BUF_IDX_57fc10a56d10, sizeof(uint64_t));
hipMemset(d_BUF_IDX_57fc10a56d10, 0, sizeof(uint64_t));
uint64_t* d_BUF_57fc10a56d10;
hipMalloc(&d_BUF_57fc10a56d10, sizeof(uint64_t) * COUNT57fc10a56d10 * 1);
auto d_HT_57fc10a56d10 = cuco::experimental::static_multimap{ (int)COUNT57fc10a56d10*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_57fc10a61860<<<std::ceil((float)part_size/32.), 32>>>(d_BUF_57fc10a56d10, d_BUF_IDX_57fc10a56d10, d_HT_57fc10a56d10.ref(cuco::insert), d_part__p_category, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT57fc10a51c00;
hipMalloc(&d_COUNT57fc10a51c00, sizeof(uint64_t));
hipMemset(d_COUNT57fc10a51c00, 0, sizeof(uint64_t));
count_57fc10a64300<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT57fc10a51c00, d_supplier__s_region, supplier_size);
uint64_t COUNT57fc10a51c00;
hipMemcpy(&COUNT57fc10a51c00, d_COUNT57fc10a51c00, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_57fc10a51c00;
hipMalloc(&d_BUF_IDX_57fc10a51c00, sizeof(uint64_t));
hipMemset(d_BUF_IDX_57fc10a51c00, 0, sizeof(uint64_t));
uint64_t* d_BUF_57fc10a51c00;
hipMalloc(&d_BUF_57fc10a51c00, sizeof(uint64_t) * COUNT57fc10a51c00 * 1);
auto d_HT_57fc10a51c00 = cuco::experimental::static_multimap{ (int)COUNT57fc10a51c00*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_57fc10a64300<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_57fc10a51c00, d_BUF_IDX_57fc10a51c00, d_HT_57fc10a51c00.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT57fc10a59000;
hipMalloc(&d_COUNT57fc10a59000, sizeof(uint64_t));
hipMemset(d_COUNT57fc10a59000, 0, sizeof(uint64_t));
count_57fc10a3a5e0<<<std::ceil((float)date_size/32.), 32>>>(d_COUNT57fc10a59000, date_size);
uint64_t COUNT57fc10a59000;
hipMemcpy(&COUNT57fc10a59000, d_COUNT57fc10a59000, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_57fc10a59000;
hipMalloc(&d_BUF_IDX_57fc10a59000, sizeof(uint64_t));
hipMemset(d_BUF_IDX_57fc10a59000, 0, sizeof(uint64_t));
uint64_t* d_BUF_57fc10a59000;
hipMalloc(&d_BUF_57fc10a59000, sizeof(uint64_t) * COUNT57fc10a59000 * 1);
auto d_HT_57fc10a59000 = cuco::experimental::static_multimap{ (int)COUNT57fc10a59000*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_57fc10a3a5e0<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_57fc10a59000, d_BUF_IDX_57fc10a59000, d_HT_57fc10a59000.ref(cuco::insert), d_date__d_datekey, date_size);
//Create aggregation hash table
auto d_HT_57fc10a0b2c0 = cuco::static_map{ (int)52974*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_57fc10a3a010<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_57fc10a51c00, d_BUF_57fc10a56d10, d_BUF_57fc10a59000, d_HT_57fc10a0b2c0.ref(cuco::insert), d_HT_57fc10a51c00.ref(cuco::for_each), d_HT_57fc10a56d10.ref(cuco::for_each), d_HT_57fc10a59000.ref(cuco::for_each), d_date__d_year, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size, d_part__p_brand1_encoded);
size_t COUNT57fc10a0b2c0 = d_HT_57fc10a0b2c0.size();
thrust::device_vector<int64_t> keys_57fc10a0b2c0(COUNT57fc10a0b2c0), vals_57fc10a0b2c0(COUNT57fc10a0b2c0);
d_HT_57fc10a0b2c0.retrieve_all(keys_57fc10a0b2c0.begin(), vals_57fc10a0b2c0.begin());
d_HT_57fc10a0b2c0.clear();
int64_t* raw_keys57fc10a0b2c0 = thrust::raw_pointer_cast(keys_57fc10a0b2c0.data());
insertKeys<<<std::ceil((float)COUNT57fc10a0b2c0/32.), 32>>>(raw_keys57fc10a0b2c0, d_HT_57fc10a0b2c0.ref(cuco::insert), COUNT57fc10a0b2c0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT57fc10a0b2c0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT57fc10a0b2c0);
DBI32Type* d_KEY_57fc10a0b2c0date__d_year;
hipMalloc(&d_KEY_57fc10a0b2c0date__d_year, sizeof(DBI32Type) * COUNT57fc10a0b2c0);
hipMemset(d_KEY_57fc10a0b2c0date__d_year, 0, sizeof(DBI32Type) * COUNT57fc10a0b2c0);
DBI16Type* d_KEY_57fc10a0b2c0part__p_brand1_encoded;
hipMalloc(&d_KEY_57fc10a0b2c0part__p_brand1_encoded, sizeof(DBI16Type) * COUNT57fc10a0b2c0);
hipMemset(d_KEY_57fc10a0b2c0part__p_brand1_encoded, 0, sizeof(DBI16Type) * COUNT57fc10a0b2c0);
main_57fc10a3a010<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_57fc10a51c00, d_BUF_57fc10a56d10, d_BUF_57fc10a59000, d_HT_57fc10a0b2c0.ref(cuco::find), d_HT_57fc10a51c00.ref(cuco::for_each), d_HT_57fc10a56d10.ref(cuco::for_each), d_HT_57fc10a59000.ref(cuco::for_each), d_KEY_57fc10a0b2c0date__d_year, d_KEY_57fc10a0b2c0part__p_brand1_encoded, d_aggr0__tmp_attr0, d_date__d_year, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_part__p_brand1_encoded);
//Materialize count
uint64_t* d_COUNT57fc10a1e470;
hipMalloc(&d_COUNT57fc10a1e470, sizeof(uint64_t));
hipMemset(d_COUNT57fc10a1e470, 0, sizeof(uint64_t));
count_57fc10a73600<<<std::ceil((float)COUNT57fc10a0b2c0/32.), 32>>>(COUNT57fc10a0b2c0, d_COUNT57fc10a1e470);
uint64_t COUNT57fc10a1e470;
hipMemcpy(&COUNT57fc10a1e470, d_COUNT57fc10a1e470, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX57fc10a1e470;
hipMalloc(&d_MAT_IDX57fc10a1e470, sizeof(uint64_t));
hipMemset(d_MAT_IDX57fc10a1e470, 0, sizeof(uint64_t));
auto MAT57fc10a1e470aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT57fc10a1e470);
DBDecimalType* d_MAT57fc10a1e470aggr0__tmp_attr0;
hipMalloc(&d_MAT57fc10a1e470aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT57fc10a1e470);
auto MAT57fc10a1e470date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT57fc10a1e470);
DBI32Type* d_MAT57fc10a1e470date__d_year;
hipMalloc(&d_MAT57fc10a1e470date__d_year, sizeof(DBI32Type) * COUNT57fc10a1e470);
auto MAT57fc10a1e470part__p_brand1_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT57fc10a1e470);
DBI16Type* d_MAT57fc10a1e470part__p_brand1_encoded;
hipMalloc(&d_MAT57fc10a1e470part__p_brand1_encoded, sizeof(DBI16Type) * COUNT57fc10a1e470);
main_57fc10a73600<<<std::ceil((float)COUNT57fc10a0b2c0/32.), 32>>>(COUNT57fc10a0b2c0, d_MAT57fc10a1e470aggr0__tmp_attr0, d_MAT57fc10a1e470date__d_year, d_MAT57fc10a1e470part__p_brand1_encoded, d_MAT_IDX57fc10a1e470, d_aggr0__tmp_attr0, d_KEY_57fc10a0b2c0date__d_year, d_KEY_57fc10a0b2c0part__p_brand1_encoded);
hipMemcpy(MAT57fc10a1e470aggr0__tmp_attr0, d_MAT57fc10a1e470aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT57fc10a1e470, hipMemcpyDeviceToHost);
hipMemcpy(MAT57fc10a1e470date__d_year, d_MAT57fc10a1e470date__d_year, sizeof(DBI32Type) * COUNT57fc10a1e470, hipMemcpyDeviceToHost);
hipMemcpy(MAT57fc10a1e470part__p_brand1_encoded, d_MAT57fc10a1e470part__p_brand1_encoded, sizeof(DBI16Type) * COUNT57fc10a1e470, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT57fc10a1e470; i++) { std::cout << MAT57fc10a1e470aggr0__tmp_attr0[i] << "\t";
std::cout << MAT57fc10a1e470date__d_year[i] << "\t";
std::cout << part__p_brand1_map[MAT57fc10a1e470part__p_brand1_encoded[i]] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_57fc10a56d10);
hipFree(d_BUF_IDX_57fc10a56d10);
hipFree(d_COUNT57fc10a56d10);
hipFree(d_BUF_57fc10a51c00);
hipFree(d_BUF_IDX_57fc10a51c00);
hipFree(d_COUNT57fc10a51c00);
hipFree(d_BUF_57fc10a59000);
hipFree(d_BUF_IDX_57fc10a59000);
hipFree(d_COUNT57fc10a59000);
hipFree(d_KEY_57fc10a0b2c0date__d_year);
hipFree(d_KEY_57fc10a0b2c0part__p_brand1_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT57fc10a1e470);
hipFree(d_MAT57fc10a1e470aggr0__tmp_attr0);
hipFree(d_MAT57fc10a1e470date__d_year);
hipFree(d_MAT57fc10a1e470part__p_brand1_encoded);
hipFree(d_MAT_IDX57fc10a1e470);
free(MAT57fc10a1e470aggr0__tmp_attr0);
free(MAT57fc10a1e470date__d_year);
free(MAT57fc10a1e470part__p_brand1_encoded);
}