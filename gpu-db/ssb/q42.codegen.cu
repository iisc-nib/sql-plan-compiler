#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_599750539bd0(uint64_t* COUNT59975052adf0, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT59975052adf0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_599750539bd0(uint64_t* BUF_59975052adf0, uint64_t* BUF_IDX_59975052adf0, HASHTABLE_INSERT HT_59975052adf0, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_59975052adf0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_59975052adf0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_59975052adf0 = atomicAdd((int*)BUF_IDX_59975052adf0, 1);
HT_59975052adf0.insert(cuco::pair{KEY_59975052adf0, buf_idx_59975052adf0});
BUF_59975052adf0[buf_idx_59975052adf0 * 1 + 0] = tid;
}
__global__ void count_5997504d1a90(uint64_t* COUNT59975052a680, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT59975052a680, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5997504d1a90(uint64_t* BUF_59975052a680, uint64_t* BUF_IDX_59975052a680, HASHTABLE_INSERT HT_59975052a680, DBI32Type* customer__c_custkey, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_59975052a680 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_59975052a680 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_59975052a680 = atomicAdd((int*)BUF_IDX_59975052a680, 1);
HT_59975052a680.insert(cuco::pair{KEY_59975052a680, buf_idx_59975052a680});
BUF_59975052a680[buf_idx_59975052a680 * 1 + 0] = tid;
}
__global__ void count_5997504d1cd0(uint64_t* COUNT59975052aba0, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT59975052aba0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5997504d1cd0(uint64_t* BUF_59975052aba0, uint64_t* BUF_IDX_59975052aba0, HASHTABLE_INSERT HT_59975052aba0, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_59975052aba0 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_59975052aba0 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_59975052aba0 = atomicAdd((int*)BUF_IDX_59975052aba0, 1);
HT_59975052aba0.insert(cuco::pair{KEY_59975052aba0, buf_idx_59975052aba0});
BUF_59975052aba0[buf_idx_59975052aba0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_59975053e190(uint64_t* BUF_59975052a680, uint64_t* BUF_59975052aba0, uint64_t* BUF_59975052adf0, uint64_t* COUNT59975052dda0, HASHTABLE_PROBE HT_59975052a680, HASHTABLE_PROBE HT_59975052aba0, HASHTABLE_PROBE HT_59975052adf0, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_59975052adf0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_59975052adf0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_59975052adf0.for_each(KEY_59975052adf0, [&] __device__ (auto const SLOT_59975052adf0) {

auto const [slot_first59975052adf0, slot_second59975052adf0] = SLOT_59975052adf0;
if (!(true)) return;
uint64_t KEY_59975052a680 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_59975052a680 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_59975052a680.for_each(KEY_59975052a680, [&] __device__ (auto const SLOT_59975052a680) {

auto const [slot_first59975052a680, slot_second59975052a680] = SLOT_59975052a680;
if (!(true)) return;
uint64_t KEY_59975052aba0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_59975052aba0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_59975052aba0.for_each(KEY_59975052aba0, [&] __device__ (auto const SLOT_59975052aba0) {

auto const [slot_first59975052aba0, slot_second59975052aba0] = SLOT_59975052aba0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT59975052dda0, 1);
});
});
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_59975053e190(uint64_t* BUF_59975052a680, uint64_t* BUF_59975052aba0, uint64_t* BUF_59975052adf0, uint64_t* BUF_59975052dda0, uint64_t* BUF_IDX_59975052dda0, HASHTABLE_PROBE HT_59975052a680, HASHTABLE_PROBE HT_59975052aba0, HASHTABLE_PROBE HT_59975052adf0, HASHTABLE_INSERT HT_59975052dda0, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_59975052adf0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_59975052adf0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_59975052adf0.for_each(KEY_59975052adf0, [&] __device__ (auto const SLOT_59975052adf0) {
auto const [slot_first59975052adf0, slot_second59975052adf0] = SLOT_59975052adf0;
if (!(true)) return;
uint64_t KEY_59975052a680 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_59975052a680 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_59975052a680.for_each(KEY_59975052a680, [&] __device__ (auto const SLOT_59975052a680) {
auto const [slot_first59975052a680, slot_second59975052a680] = SLOT_59975052a680;
if (!(true)) return;
uint64_t KEY_59975052aba0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_59975052aba0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_59975052aba0.for_each(KEY_59975052aba0, [&] __device__ (auto const SLOT_59975052aba0) {
auto const [slot_first59975052aba0, slot_second59975052aba0] = SLOT_59975052aba0;
if (!(true)) return;
uint64_t KEY_59975052dda0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_59975052dda0 |= reg_lineorder__lo_partkey;
// Insert hash table kernel;
auto buf_idx_59975052dda0 = atomicAdd((int*)BUF_IDX_59975052dda0, 1);
HT_59975052dda0.insert(cuco::pair{KEY_59975052dda0, buf_idx_59975052dda0});
BUF_59975052dda0[buf_idx_59975052dda0 * 4 + 0] = BUF_59975052aba0[slot_second59975052aba0 * 1 + 0];
BUF_59975052dda0[buf_idx_59975052dda0 * 4 + 1] = BUF_59975052a680[slot_second59975052a680 * 1 + 0];
BUF_59975052dda0[buf_idx_59975052dda0 * 4 + 2] = BUF_59975052adf0[slot_second59975052adf0 * 1 + 0];
BUF_59975052dda0[buf_idx_59975052dda0 * 4 + 3] = tid;
});
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_59975053bee0(uint64_t* BUF_59975052dda0, HASHTABLE_INSERT HT_5997504e1a50, HASHTABLE_PROBE HT_59975052dda0, DBI32Type* date__d_year, DBI16Type* part__p_category_encoded, DBStringType* part__p_mfgr, DBI32Type* part__p_partkey, size_t part_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_mfgr = part__p_mfgr[tid];
if (!((evaluatePredicate(reg_part__p_mfgr, "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr, "MFGR#2", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_59975052dda0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_59975052dda0 |= reg_part__p_partkey;
//Probe Hash table
HT_59975052dda0.for_each(KEY_59975052dda0, [&] __device__ (auto const SLOT_59975052dda0) {

auto const [slot_first59975052dda0, slot_second59975052dda0] = SLOT_59975052dda0;
if (!(true)) return;
uint64_t KEY_5997504e1a50 = 0;
auto reg_date__d_year = date__d_year[BUF_59975052dda0[slot_second59975052dda0 * 4 + 0]];

KEY_5997504e1a50 |= reg_date__d_year;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_59975052dda0[slot_second59975052dda0 * 4 + 2]];
KEY_5997504e1a50 <<= 16;
KEY_5997504e1a50 |= reg_supplier__s_nation_encoded;
auto reg_part__p_category_encoded = part__p_category_encoded[tid];
KEY_5997504e1a50 <<= 16;
KEY_5997504e1a50 |= reg_part__p_category_encoded;
//Create aggregation hash table
HT_5997504e1a50.insert(cuco::pair{KEY_5997504e1a50, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_59975053bee0(uint64_t* BUF_59975052dda0, HASHTABLE_FIND HT_5997504e1a50, HASHTABLE_PROBE HT_59975052dda0, DBI32Type* KEY_5997504e1a50date__d_year, DBI16Type* KEY_5997504e1a50part__p_category_encoded, DBI16Type* KEY_5997504e1a50supplier__s_nation_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBDecimalType* lineorder__lo_revenue, DBDecimalType* lineorder__lo_supplycost, DBI16Type* part__p_category_encoded, DBStringType* part__p_mfgr, DBI32Type* part__p_partkey, size_t part_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_mfgr = part__p_mfgr[tid];
if (!((evaluatePredicate(reg_part__p_mfgr, "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr, "MFGR#2", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_59975052dda0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_59975052dda0 |= reg_part__p_partkey;
//Probe Hash table
HT_59975052dda0.for_each(KEY_59975052dda0, [&] __device__ (auto const SLOT_59975052dda0) {
auto const [slot_first59975052dda0, slot_second59975052dda0] = SLOT_59975052dda0;
if (!(true)) return;
uint64_t KEY_5997504e1a50 = 0;
auto reg_date__d_year = date__d_year[BUF_59975052dda0[slot_second59975052dda0 * 4 + 0]];

KEY_5997504e1a50 |= reg_date__d_year;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_59975052dda0[slot_second59975052dda0 * 4 + 2]];
KEY_5997504e1a50 <<= 16;
KEY_5997504e1a50 |= reg_supplier__s_nation_encoded;
auto reg_part__p_category_encoded = part__p_category_encoded[tid];
KEY_5997504e1a50 <<= 16;
KEY_5997504e1a50 |= reg_part__p_category_encoded;
//Aggregate in hashtable
auto buf_idx_5997504e1a50 = HT_5997504e1a50.find(KEY_5997504e1a50)->second;
auto reg_lineorder__lo_supplycost = lineorder__lo_supplycost[BUF_59975052dda0[slot_second59975052dda0 * 4 + 3]];
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_59975052dda0[slot_second59975052dda0 * 4 + 3]];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_revenue) - (reg_lineorder__lo_supplycost);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5997504e1a50], reg_map0__tmp_attr1);
KEY_5997504e1a50date__d_year[buf_idx_5997504e1a50] = reg_date__d_year;
KEY_5997504e1a50supplier__s_nation_encoded[buf_idx_5997504e1a50] = reg_supplier__s_nation_encoded;
KEY_5997504e1a50part__p_category_encoded[buf_idx_5997504e1a50] = reg_part__p_category_encoded;
});
}
__global__ void count_599750556750(uint64_t* COUNT5997504c0ad0, size_t COUNT5997504e1a50) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5997504e1a50) return;
//Materialize count
atomicAdd((int*)COUNT5997504c0ad0, 1);
}
__global__ void main_599750556750(size_t COUNT5997504e1a50, DBDecimalType* MAT5997504c0ad0aggr0__tmp_attr0, DBI32Type* MAT5997504c0ad0date__d_year, DBI16Type* MAT5997504c0ad0part__p_category_encoded, DBI16Type* MAT5997504c0ad0supplier__s_nation_encoded, uint64_t* MAT_IDX5997504c0ad0, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_category_encoded, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5997504e1a50) return;
//Materialize buffers
auto mat_idx5997504c0ad0 = atomicAdd((int*)MAT_IDX5997504c0ad0, 1);
auto reg_date__d_year = date__d_year[tid];
MAT5997504c0ad0date__d_year[mat_idx5997504c0ad0] = reg_date__d_year;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[tid];
MAT5997504c0ad0supplier__s_nation_encoded[mat_idx5997504c0ad0] = reg_supplier__s_nation_encoded;
auto reg_part__p_category_encoded = part__p_category_encoded[tid];
MAT5997504c0ad0part__p_category_encoded[mat_idx5997504c0ad0] = reg_part__p_category_encoded;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5997504c0ad0aggr0__tmp_attr0[mat_idx5997504c0ad0] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map) {
//Materialize count
uint64_t* d_COUNT59975052adf0;
hipMalloc(&d_COUNT59975052adf0, sizeof(uint64_t));
hipMemset(d_COUNT59975052adf0, 0, sizeof(uint64_t));
count_599750539bd0<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT59975052adf0, d_supplier__s_region, supplier_size);
uint64_t COUNT59975052adf0;
hipMemcpy(&COUNT59975052adf0, d_COUNT59975052adf0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_59975052adf0;
hipMalloc(&d_BUF_IDX_59975052adf0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_59975052adf0, 0, sizeof(uint64_t));
uint64_t* d_BUF_59975052adf0;
hipMalloc(&d_BUF_59975052adf0, sizeof(uint64_t) * COUNT59975052adf0 * 1);
auto d_HT_59975052adf0 = cuco::experimental::static_multimap{ (int)COUNT59975052adf0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_599750539bd0<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_59975052adf0, d_BUF_IDX_59975052adf0, d_HT_59975052adf0.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT59975052a680;
hipMalloc(&d_COUNT59975052a680, sizeof(uint64_t));
hipMemset(d_COUNT59975052a680, 0, sizeof(uint64_t));
count_5997504d1a90<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT59975052a680, d_customer__c_region, customer_size);
uint64_t COUNT59975052a680;
hipMemcpy(&COUNT59975052a680, d_COUNT59975052a680, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_59975052a680;
hipMalloc(&d_BUF_IDX_59975052a680, sizeof(uint64_t));
hipMemset(d_BUF_IDX_59975052a680, 0, sizeof(uint64_t));
uint64_t* d_BUF_59975052a680;
hipMalloc(&d_BUF_59975052a680, sizeof(uint64_t) * COUNT59975052a680 * 1);
auto d_HT_59975052a680 = cuco::experimental::static_multimap{ (int)COUNT59975052a680*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5997504d1a90<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_59975052a680, d_BUF_IDX_59975052a680, d_HT_59975052a680.ref(cuco::insert), d_customer__c_custkey, d_customer__c_region, customer_size);
//Materialize count
uint64_t* d_COUNT59975052aba0;
hipMalloc(&d_COUNT59975052aba0, sizeof(uint64_t));
hipMemset(d_COUNT59975052aba0, 0, sizeof(uint64_t));
count_5997504d1cd0<<<std::ceil((float)date_size/32.), 32>>>(d_COUNT59975052aba0, d_date__d_year, date_size);
uint64_t COUNT59975052aba0;
hipMemcpy(&COUNT59975052aba0, d_COUNT59975052aba0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_59975052aba0;
hipMalloc(&d_BUF_IDX_59975052aba0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_59975052aba0, 0, sizeof(uint64_t));
uint64_t* d_BUF_59975052aba0;
hipMalloc(&d_BUF_59975052aba0, sizeof(uint64_t) * COUNT59975052aba0 * 1);
auto d_HT_59975052aba0 = cuco::experimental::static_multimap{ (int)COUNT59975052aba0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5997504d1cd0<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_59975052aba0, d_BUF_IDX_59975052aba0, d_HT_59975052aba0.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Materialize count
uint64_t* d_COUNT59975052dda0;
hipMalloc(&d_COUNT59975052dda0, sizeof(uint64_t));
hipMemset(d_COUNT59975052dda0, 0, sizeof(uint64_t));
count_59975053e190<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_59975052a680, d_BUF_59975052aba0, d_BUF_59975052adf0, d_COUNT59975052dda0, d_HT_59975052a680.ref(cuco::for_each), d_HT_59975052aba0.ref(cuco::for_each), d_HT_59975052adf0.ref(cuco::for_each), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size);
uint64_t COUNT59975052dda0;
hipMemcpy(&COUNT59975052dda0, d_COUNT59975052dda0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_59975052dda0;
hipMalloc(&d_BUF_IDX_59975052dda0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_59975052dda0, 0, sizeof(uint64_t));
uint64_t* d_BUF_59975052dda0;
hipMalloc(&d_BUF_59975052dda0, sizeof(uint64_t) * COUNT59975052dda0 * 4);
auto d_HT_59975052dda0 = cuco::experimental::static_multimap{ (int)COUNT59975052dda0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_59975053e190<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_59975052a680, d_BUF_59975052aba0, d_BUF_59975052adf0, d_BUF_59975052dda0, d_BUF_IDX_59975052dda0, d_HT_59975052a680.ref(cuco::for_each), d_HT_59975052aba0.ref(cuco::for_each), d_HT_59975052adf0.ref(cuco::for_each), d_HT_59975052dda0.ref(cuco::insert), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
//Create aggregation hash table
auto d_HT_5997504e1a50 = cuco::static_map{ (int)24650*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_59975053bee0<<<std::ceil((float)part_size/32.), 32>>>(d_BUF_59975052dda0, d_HT_5997504e1a50.ref(cuco::insert), d_HT_59975052dda0.ref(cuco::for_each), d_date__d_year, , d_part__p_mfgr, d_part__p_partkey, part_size, d_supplier__s_nation_encoded);
size_t COUNT5997504e1a50 = d_HT_5997504e1a50.size();
thrust::device_vector<int64_t> keys_5997504e1a50(COUNT5997504e1a50), vals_5997504e1a50(COUNT5997504e1a50);
d_HT_5997504e1a50.retrieve_all(keys_5997504e1a50.begin(), vals_5997504e1a50.begin());
d_HT_5997504e1a50.clear();
int64_t* raw_keys5997504e1a50 = thrust::raw_pointer_cast(keys_5997504e1a50.data());
insertKeys<<<std::ceil((float)COUNT5997504e1a50/32.), 32>>>(raw_keys5997504e1a50, d_HT_5997504e1a50.ref(cuco::insert), COUNT5997504e1a50);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5997504e1a50);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5997504e1a50);
DBI32Type* d_KEY_5997504e1a50date__d_year;
hipMalloc(&d_KEY_5997504e1a50date__d_year, sizeof(DBI32Type) * COUNT5997504e1a50);
hipMemset(d_KEY_5997504e1a50date__d_year, 0, sizeof(DBI32Type) * COUNT5997504e1a50);
DBI16Type* d_KEY_5997504e1a50supplier__s_nation_encoded;
hipMalloc(&d_KEY_5997504e1a50supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT5997504e1a50);
hipMemset(d_KEY_5997504e1a50supplier__s_nation_encoded, 0, sizeof(DBI16Type) * COUNT5997504e1a50);
DBI16Type* d_KEY_5997504e1a50part__p_category_encoded;
hipMalloc(&d_KEY_5997504e1a50part__p_category_encoded, sizeof(DBI16Type) * COUNT5997504e1a50);
hipMemset(d_KEY_5997504e1a50part__p_category_encoded, 0, sizeof(DBI16Type) * COUNT5997504e1a50);
main_59975053bee0<<<std::ceil((float)part_size/32.), 32>>>(d_BUF_59975052dda0, d_HT_5997504e1a50.ref(cuco::find), d_HT_59975052dda0.ref(cuco::for_each), d_KEY_5997504e1a50date__d_year, d_KEY_5997504e1a50part__p_category_encoded, d_KEY_5997504e1a50supplier__s_nation_encoded, d_aggr0__tmp_attr0, d_date__d_year, d_lineorder__lo_revenue, d_lineorder__lo_supplycost, , d_part__p_mfgr, d_part__p_partkey, part_size, d_supplier__s_nation_encoded);
//Materialize count
uint64_t* d_COUNT5997504c0ad0;
hipMalloc(&d_COUNT5997504c0ad0, sizeof(uint64_t));
hipMemset(d_COUNT5997504c0ad0, 0, sizeof(uint64_t));
count_599750556750<<<std::ceil((float)COUNT5997504e1a50/32.), 32>>>(d_COUNT5997504c0ad0, COUNT5997504e1a50);
uint64_t COUNT5997504c0ad0;
hipMemcpy(&COUNT5997504c0ad0, d_COUNT5997504c0ad0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5997504c0ad0;
hipMalloc(&d_MAT_IDX5997504c0ad0, sizeof(uint64_t));
hipMemset(d_MAT_IDX5997504c0ad0, 0, sizeof(uint64_t));
auto MAT5997504c0ad0date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5997504c0ad0);
DBI32Type* d_MAT5997504c0ad0date__d_year;
hipMalloc(&d_MAT5997504c0ad0date__d_year, sizeof(DBI32Type) * COUNT5997504c0ad0);
auto MAT5997504c0ad0supplier__s_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5997504c0ad0);
DBI16Type* d_MAT5997504c0ad0supplier__s_nation_encoded;
hipMalloc(&d_MAT5997504c0ad0supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT5997504c0ad0);
auto MAT5997504c0ad0part__p_category_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5997504c0ad0);
DBI16Type* d_MAT5997504c0ad0part__p_category_encoded;
hipMalloc(&d_MAT5997504c0ad0part__p_category_encoded, sizeof(DBI16Type) * COUNT5997504c0ad0);
auto MAT5997504c0ad0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5997504c0ad0);
DBDecimalType* d_MAT5997504c0ad0aggr0__tmp_attr0;
hipMalloc(&d_MAT5997504c0ad0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5997504c0ad0);
main_599750556750<<<std::ceil((float)COUNT5997504e1a50/32.), 32>>>(COUNT5997504e1a50, d_MAT5997504c0ad0aggr0__tmp_attr0, d_MAT5997504c0ad0date__d_year, d_MAT5997504c0ad0part__p_category_encoded, d_MAT5997504c0ad0supplier__s_nation_encoded, d_MAT_IDX5997504c0ad0, d_aggr0__tmp_attr0, d_KEY_5997504e1a50date__d_year, d_KEY_5997504e1a50part__p_category_encoded, d_KEY_5997504e1a50supplier__s_nation_encoded);
hipMemcpy(MAT5997504c0ad0date__d_year, d_MAT5997504c0ad0date__d_year, sizeof(DBI32Type) * COUNT5997504c0ad0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5997504c0ad0supplier__s_nation_encoded, d_MAT5997504c0ad0supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT5997504c0ad0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5997504c0ad0part__p_category_encoded, d_MAT5997504c0ad0part__p_category_encoded, sizeof(DBI16Type) * COUNT5997504c0ad0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5997504c0ad0aggr0__tmp_attr0, d_MAT5997504c0ad0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5997504c0ad0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5997504c0ad0; i++) { std::cout << MAT5997504c0ad0date__d_year[i] << "\t";
std::cout << supplier__s_nation_map[MAT5997504c0ad0supplier__s_nation_encoded[i]] << "\t";
std::cout << part__p_category_map[MAT5997504c0ad0part__p_category_encoded[i]] << "\t";
std::cout << MAT5997504c0ad0aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_59975052adf0);
hipFree(d_BUF_IDX_59975052adf0);
hipFree(d_COUNT59975052adf0);
hipFree(d_BUF_59975052a680);
hipFree(d_BUF_IDX_59975052a680);
hipFree(d_COUNT59975052a680);
hipFree(d_BUF_59975052aba0);
hipFree(d_BUF_IDX_59975052aba0);
hipFree(d_COUNT59975052aba0);
hipFree(d_BUF_59975052dda0);
hipFree(d_BUF_IDX_59975052dda0);
hipFree(d_COUNT59975052dda0);
hipFree(d_KEY_5997504e1a50date__d_year);
hipFree(d_KEY_5997504e1a50part__p_category_encoded);
hipFree(d_KEY_5997504e1a50supplier__s_nation_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5997504c0ad0);
hipFree(d_MAT5997504c0ad0aggr0__tmp_attr0);
hipFree(d_MAT5997504c0ad0date__d_year);
hipFree(d_MAT5997504c0ad0part__p_category_encoded);
hipFree(d_MAT5997504c0ad0supplier__s_nation_encoded);
hipFree(d_MAT_IDX5997504c0ad0);
free(MAT5997504c0ad0aggr0__tmp_attr0);
free(MAT5997504c0ad0date__d_year);
free(MAT5997504c0ad0part__p_category_encoded);
free(MAT5997504c0ad0supplier__s_nation_encoded);
}