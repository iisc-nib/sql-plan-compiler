#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_61b1de23ce00(uint64_t* COUNT61b1de22db60, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT61b1de22db60, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_61b1de23ce00(uint64_t* BUF_61b1de22db60, uint64_t* BUF_IDX_61b1de22db60, HASHTABLE_INSERT HT_61b1de22db60, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_61b1de22db60 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_61b1de22db60 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_61b1de22db60 = atomicAdd((int*)BUF_IDX_61b1de22db60, 1);
HT_61b1de22db60.insert(cuco::pair{KEY_61b1de22db60, buf_idx_61b1de22db60});
BUF_61b1de22db60[buf_idx_61b1de22db60 * 1 + 0] = tid;
}
__global__ void count_61b1de1d40d0(uint64_t* COUNT61b1de22dc90, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT61b1de22dc90, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_61b1de1d40d0(uint64_t* BUF_61b1de22dc90, uint64_t* BUF_IDX_61b1de22dc90, HASHTABLE_INSERT HT_61b1de22dc90, DBI32Type* customer__c_custkey, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_61b1de22dc90 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_61b1de22dc90 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_61b1de22dc90 = atomicAdd((int*)BUF_IDX_61b1de22dc90, 1);
HT_61b1de22dc90.insert(cuco::pair{KEY_61b1de22dc90, buf_idx_61b1de22dc90});
BUF_61b1de22dc90[buf_idx_61b1de22dc90 * 1 + 0] = tid;
}
__global__ void count_61b1de1d4310(uint64_t* COUNT61b1de231120, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT61b1de231120, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_61b1de1d4310(uint64_t* BUF_61b1de231120, uint64_t* BUF_IDX_61b1de231120, HASHTABLE_INSERT HT_61b1de231120, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_61b1de231120 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_61b1de231120 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_61b1de231120 = atomicAdd((int*)BUF_IDX_61b1de231120, 1);
HT_61b1de231120.insert(cuco::pair{KEY_61b1de231120, buf_idx_61b1de231120});
BUF_61b1de231120[buf_idx_61b1de231120 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_61b1de241e80(uint64_t* BUF_61b1de22db60, uint64_t* BUF_61b1de22dc90, uint64_t* BUF_61b1de231120, uint64_t* COUNT61b1de20e760, HASHTABLE_PROBE HT_61b1de22db60, HASHTABLE_PROBE HT_61b1de22dc90, HASHTABLE_PROBE HT_61b1de231120, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_61b1de22db60 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_61b1de22db60 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_61b1de22db60.for_each(KEY_61b1de22db60, [&] __device__ (auto const SLOT_61b1de22db60) {

auto const [slot_first61b1de22db60, slot_second61b1de22db60] = SLOT_61b1de22db60;
if (!(true)) return;
uint64_t KEY_61b1de22dc90 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_61b1de22dc90 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_61b1de22dc90.for_each(KEY_61b1de22dc90, [&] __device__ (auto const SLOT_61b1de22dc90) {

auto const [slot_first61b1de22dc90, slot_second61b1de22dc90] = SLOT_61b1de22dc90;
if (!(true)) return;
uint64_t KEY_61b1de231120 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_61b1de231120 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_61b1de231120.for_each(KEY_61b1de231120, [&] __device__ (auto const SLOT_61b1de231120) {

auto const [slot_first61b1de231120, slot_second61b1de231120] = SLOT_61b1de231120;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT61b1de20e760, 1);
});
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_61b1de241e80(uint64_t* BUF_61b1de20e760, uint64_t* BUF_61b1de22db60, uint64_t* BUF_61b1de22dc90, uint64_t* BUF_61b1de231120, uint64_t* BUF_IDX_61b1de20e760, HASHTABLE_INSERT HT_61b1de20e760, HASHTABLE_PROBE HT_61b1de22db60, HASHTABLE_PROBE HT_61b1de22dc90, HASHTABLE_PROBE HT_61b1de231120, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_61b1de22db60 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_61b1de22db60 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_61b1de22db60.for_each(KEY_61b1de22db60, [&] __device__ (auto const SLOT_61b1de22db60) {
auto const [slot_first61b1de22db60, slot_second61b1de22db60] = SLOT_61b1de22db60;
if (!(true)) return;
uint64_t KEY_61b1de22dc90 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_61b1de22dc90 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_61b1de22dc90.for_each(KEY_61b1de22dc90, [&] __device__ (auto const SLOT_61b1de22dc90) {
auto const [slot_first61b1de22dc90, slot_second61b1de22dc90] = SLOT_61b1de22dc90;
if (!(true)) return;
uint64_t KEY_61b1de231120 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_61b1de231120 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_61b1de231120.for_each(KEY_61b1de231120, [&] __device__ (auto const SLOT_61b1de231120) {
auto const [slot_first61b1de231120, slot_second61b1de231120] = SLOT_61b1de231120;
if (!(true)) return;
uint64_t KEY_61b1de20e760 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_61b1de20e760 |= reg_lineorder__lo_partkey;
// Insert hash table kernel;
auto buf_idx_61b1de20e760 = atomicAdd((int*)BUF_IDX_61b1de20e760, 1);
HT_61b1de20e760.insert(cuco::pair{KEY_61b1de20e760, buf_idx_61b1de20e760});
BUF_61b1de20e760[buf_idx_61b1de20e760 * 4 + 0] = BUF_61b1de231120[slot_second61b1de231120 * 1 + 0];
BUF_61b1de20e760[buf_idx_61b1de20e760 * 4 + 1] = BUF_61b1de22dc90[slot_second61b1de22dc90 * 1 + 0];
BUF_61b1de20e760[buf_idx_61b1de20e760 * 4 + 2] = BUF_61b1de22db60[slot_second61b1de22db60 * 1 + 0];
BUF_61b1de20e760[buf_idx_61b1de20e760 * 4 + 3] = tid;
});
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_61b1de23f620(uint64_t* BUF_61b1de20e760, HASHTABLE_INSERT HT_61b1de1e4150, HASHTABLE_PROBE HT_61b1de20e760, DBI32Type* date__d_year, DBI16Type* part__p_category_encoded, DBStringType* part__p_mfgr, DBI32Type* part__p_partkey, size_t part_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_mfgr = part__p_mfgr[tid];
if (!((evaluatePredicate(reg_part__p_mfgr, "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr, "MFGR#2", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_61b1de20e760 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_61b1de20e760 |= reg_part__p_partkey;
//Probe Hash table
HT_61b1de20e760.for_each(KEY_61b1de20e760, [&] __device__ (auto const SLOT_61b1de20e760) {

auto const [slot_first61b1de20e760, slot_second61b1de20e760] = SLOT_61b1de20e760;
if (!(true)) return;
uint64_t KEY_61b1de1e4150 = 0;
auto reg_date__d_year = date__d_year[BUF_61b1de20e760[slot_second61b1de20e760 * 4 + 0]];

KEY_61b1de1e4150 |= reg_date__d_year;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_61b1de20e760[slot_second61b1de20e760 * 4 + 2]];
KEY_61b1de1e4150 <<= 16;
KEY_61b1de1e4150 |= reg_supplier__s_nation_encoded;
auto reg_part__p_category_encoded = part__p_category_encoded[tid];
KEY_61b1de1e4150 <<= 16;
KEY_61b1de1e4150 |= reg_part__p_category_encoded;
//Create aggregation hash table
HT_61b1de1e4150.insert(cuco::pair{KEY_61b1de1e4150, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_61b1de23f620(uint64_t* BUF_61b1de20e760, HASHTABLE_FIND HT_61b1de1e4150, HASHTABLE_PROBE HT_61b1de20e760, DBI32Type* KEY_61b1de1e4150date__d_year, DBI16Type* KEY_61b1de1e4150part__p_category_encoded, DBI16Type* KEY_61b1de1e4150supplier__s_nation_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBDecimalType* lineorder__lo_revenue, DBDecimalType* lineorder__lo_supplycost, DBI16Type* part__p_category_encoded, DBStringType* part__p_mfgr, DBI32Type* part__p_partkey, size_t part_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_mfgr = part__p_mfgr[tid];
if (!((evaluatePredicate(reg_part__p_mfgr, "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr, "MFGR#2", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_61b1de20e760 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_61b1de20e760 |= reg_part__p_partkey;
//Probe Hash table
HT_61b1de20e760.for_each(KEY_61b1de20e760, [&] __device__ (auto const SLOT_61b1de20e760) {
auto const [slot_first61b1de20e760, slot_second61b1de20e760] = SLOT_61b1de20e760;
if (!(true)) return;
uint64_t KEY_61b1de1e4150 = 0;
auto reg_date__d_year = date__d_year[BUF_61b1de20e760[slot_second61b1de20e760 * 4 + 0]];

KEY_61b1de1e4150 |= reg_date__d_year;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_61b1de20e760[slot_second61b1de20e760 * 4 + 2]];
KEY_61b1de1e4150 <<= 16;
KEY_61b1de1e4150 |= reg_supplier__s_nation_encoded;
auto reg_part__p_category_encoded = part__p_category_encoded[tid];
KEY_61b1de1e4150 <<= 16;
KEY_61b1de1e4150 |= reg_part__p_category_encoded;
//Aggregate in hashtable
auto buf_idx_61b1de1e4150 = HT_61b1de1e4150.find(KEY_61b1de1e4150)->second;
auto reg_lineorder__lo_supplycost = lineorder__lo_supplycost[BUF_61b1de20e760[slot_second61b1de20e760 * 4 + 3]];
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_61b1de20e760[slot_second61b1de20e760 * 4 + 3]];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_revenue) - (reg_lineorder__lo_supplycost);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_61b1de1e4150], reg_map0__tmp_attr1);
KEY_61b1de1e4150date__d_year[buf_idx_61b1de1e4150] = reg_date__d_year;
KEY_61b1de1e4150supplier__s_nation_encoded[buf_idx_61b1de1e4150] = reg_supplier__s_nation_encoded;
KEY_61b1de1e4150part__p_category_encoded[buf_idx_61b1de1e4150] = reg_part__p_category_encoded;
});
}
__global__ void count_61b1de2520c0(uint64_t* COUNT61b1de1c3110, size_t COUNT61b1de1e4150) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT61b1de1e4150) return;
//Materialize count
atomicAdd((int*)COUNT61b1de1c3110, 1);
}
__global__ void main_61b1de2520c0(size_t COUNT61b1de1e4150, DBDecimalType* MAT61b1de1c3110aggr0__tmp_attr0, DBI32Type* MAT61b1de1c3110date__d_year, DBI16Type* MAT61b1de1c3110part__p_category_encoded, DBI16Type* MAT61b1de1c3110supplier__s_nation_encoded, uint64_t* MAT_IDX61b1de1c3110, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_category_encoded, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT61b1de1e4150) return;
//Materialize buffers
auto mat_idx61b1de1c3110 = atomicAdd((int*)MAT_IDX61b1de1c3110, 1);
auto reg_date__d_year = date__d_year[tid];
MAT61b1de1c3110date__d_year[mat_idx61b1de1c3110] = reg_date__d_year;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[tid];
MAT61b1de1c3110supplier__s_nation_encoded[mat_idx61b1de1c3110] = reg_supplier__s_nation_encoded;
auto reg_part__p_category_encoded = part__p_category_encoded[tid];
MAT61b1de1c3110part__p_category_encoded[mat_idx61b1de1c3110] = reg_part__p_category_encoded;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT61b1de1c3110aggr0__tmp_attr0[mat_idx61b1de1c3110] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT61b1de22db60;
hipMalloc(&d_COUNT61b1de22db60, sizeof(uint64_t));
hipMemset(d_COUNT61b1de22db60, 0, sizeof(uint64_t));
count_61b1de23ce00<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT61b1de22db60, d_supplier__s_region, supplier_size);
uint64_t COUNT61b1de22db60;
hipMemcpy(&COUNT61b1de22db60, d_COUNT61b1de22db60, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_61b1de22db60;
hipMalloc(&d_BUF_IDX_61b1de22db60, sizeof(uint64_t));
hipMemset(d_BUF_IDX_61b1de22db60, 0, sizeof(uint64_t));
uint64_t* d_BUF_61b1de22db60;
hipMalloc(&d_BUF_61b1de22db60, sizeof(uint64_t) * COUNT61b1de22db60 * 1);
auto d_HT_61b1de22db60 = cuco::experimental::static_multimap{ (int)COUNT61b1de22db60*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_61b1de23ce00<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_61b1de22db60, d_BUF_IDX_61b1de22db60, d_HT_61b1de22db60.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT61b1de22dc90;
hipMalloc(&d_COUNT61b1de22dc90, sizeof(uint64_t));
hipMemset(d_COUNT61b1de22dc90, 0, sizeof(uint64_t));
count_61b1de1d40d0<<<std::ceil((float)customer_size/128.), 128>>>(d_COUNT61b1de22dc90, d_customer__c_region, customer_size);
uint64_t COUNT61b1de22dc90;
hipMemcpy(&COUNT61b1de22dc90, d_COUNT61b1de22dc90, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_61b1de22dc90;
hipMalloc(&d_BUF_IDX_61b1de22dc90, sizeof(uint64_t));
hipMemset(d_BUF_IDX_61b1de22dc90, 0, sizeof(uint64_t));
uint64_t* d_BUF_61b1de22dc90;
hipMalloc(&d_BUF_61b1de22dc90, sizeof(uint64_t) * COUNT61b1de22dc90 * 1);
auto d_HT_61b1de22dc90 = cuco::experimental::static_multimap{ (int)COUNT61b1de22dc90*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_61b1de1d40d0<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_61b1de22dc90, d_BUF_IDX_61b1de22dc90, d_HT_61b1de22dc90.ref(cuco::insert), d_customer__c_custkey, d_customer__c_region, customer_size);
//Materialize count
uint64_t* d_COUNT61b1de231120;
hipMalloc(&d_COUNT61b1de231120, sizeof(uint64_t));
hipMemset(d_COUNT61b1de231120, 0, sizeof(uint64_t));
count_61b1de1d4310<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT61b1de231120, d_date__d_year, date_size);
uint64_t COUNT61b1de231120;
hipMemcpy(&COUNT61b1de231120, d_COUNT61b1de231120, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_61b1de231120;
hipMalloc(&d_BUF_IDX_61b1de231120, sizeof(uint64_t));
hipMemset(d_BUF_IDX_61b1de231120, 0, sizeof(uint64_t));
uint64_t* d_BUF_61b1de231120;
hipMalloc(&d_BUF_61b1de231120, sizeof(uint64_t) * COUNT61b1de231120 * 1);
auto d_HT_61b1de231120 = cuco::experimental::static_multimap{ (int)COUNT61b1de231120*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_61b1de1d4310<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_61b1de231120, d_BUF_IDX_61b1de231120, d_HT_61b1de231120.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Materialize count
uint64_t* d_COUNT61b1de20e760;
hipMalloc(&d_COUNT61b1de20e760, sizeof(uint64_t));
hipMemset(d_COUNT61b1de20e760, 0, sizeof(uint64_t));
count_61b1de241e80<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_61b1de22db60, d_BUF_61b1de22dc90, d_BUF_61b1de231120, d_COUNT61b1de20e760, d_HT_61b1de22db60.ref(cuco::for_each), d_HT_61b1de22dc90.ref(cuco::for_each), d_HT_61b1de231120.ref(cuco::for_each), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size);
uint64_t COUNT61b1de20e760;
hipMemcpy(&COUNT61b1de20e760, d_COUNT61b1de20e760, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_61b1de20e760;
hipMalloc(&d_BUF_IDX_61b1de20e760, sizeof(uint64_t));
hipMemset(d_BUF_IDX_61b1de20e760, 0, sizeof(uint64_t));
uint64_t* d_BUF_61b1de20e760;
hipMalloc(&d_BUF_61b1de20e760, sizeof(uint64_t) * COUNT61b1de20e760 * 4);
auto d_HT_61b1de20e760 = cuco::experimental::static_multimap{ (int)COUNT61b1de20e760*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_61b1de241e80<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_61b1de20e760, d_BUF_61b1de22db60, d_BUF_61b1de22dc90, d_BUF_61b1de231120, d_BUF_IDX_61b1de20e760, d_HT_61b1de20e760.ref(cuco::insert), d_HT_61b1de22db60.ref(cuco::for_each), d_HT_61b1de22dc90.ref(cuco::for_each), d_HT_61b1de231120.ref(cuco::for_each), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
//Create aggregation hash table
auto d_HT_61b1de1e4150 = cuco::static_map{ (int)24650*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_61b1de23f620<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_61b1de20e760, d_HT_61b1de1e4150.ref(cuco::insert), d_HT_61b1de20e760.ref(cuco::for_each), d_date__d_year, d_part__p_category_encoded, d_part__p_mfgr, d_part__p_partkey, part_size, d_supplier__s_nation_encoded);
size_t COUNT61b1de1e4150 = d_HT_61b1de1e4150.size();
thrust::device_vector<int64_t> keys_61b1de1e4150(COUNT61b1de1e4150), vals_61b1de1e4150(COUNT61b1de1e4150);
d_HT_61b1de1e4150.retrieve_all(keys_61b1de1e4150.begin(), vals_61b1de1e4150.begin());
d_HT_61b1de1e4150.clear();
int64_t* raw_keys61b1de1e4150 = thrust::raw_pointer_cast(keys_61b1de1e4150.data());
insertKeys<<<std::ceil((float)COUNT61b1de1e4150/128.), 128>>>(raw_keys61b1de1e4150, d_HT_61b1de1e4150.ref(cuco::insert), COUNT61b1de1e4150);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT61b1de1e4150);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT61b1de1e4150);
DBI32Type* d_KEY_61b1de1e4150date__d_year;
hipMalloc(&d_KEY_61b1de1e4150date__d_year, sizeof(DBI32Type) * COUNT61b1de1e4150);
hipMemset(d_KEY_61b1de1e4150date__d_year, 0, sizeof(DBI32Type) * COUNT61b1de1e4150);
DBI16Type* d_KEY_61b1de1e4150supplier__s_nation_encoded;
hipMalloc(&d_KEY_61b1de1e4150supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT61b1de1e4150);
hipMemset(d_KEY_61b1de1e4150supplier__s_nation_encoded, 0, sizeof(DBI16Type) * COUNT61b1de1e4150);
DBI16Type* d_KEY_61b1de1e4150part__p_category_encoded;
hipMalloc(&d_KEY_61b1de1e4150part__p_category_encoded, sizeof(DBI16Type) * COUNT61b1de1e4150);
hipMemset(d_KEY_61b1de1e4150part__p_category_encoded, 0, sizeof(DBI16Type) * COUNT61b1de1e4150);
main_61b1de23f620<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_61b1de20e760, d_HT_61b1de1e4150.ref(cuco::find), d_HT_61b1de20e760.ref(cuco::for_each), d_KEY_61b1de1e4150date__d_year, d_KEY_61b1de1e4150part__p_category_encoded, d_KEY_61b1de1e4150supplier__s_nation_encoded, d_aggr0__tmp_attr0, d_date__d_year, d_lineorder__lo_revenue, d_lineorder__lo_supplycost, d_part__p_category_encoded, d_part__p_mfgr, d_part__p_partkey, part_size, d_supplier__s_nation_encoded);
//Materialize count
uint64_t* d_COUNT61b1de1c3110;
hipMalloc(&d_COUNT61b1de1c3110, sizeof(uint64_t));
hipMemset(d_COUNT61b1de1c3110, 0, sizeof(uint64_t));
count_61b1de2520c0<<<std::ceil((float)COUNT61b1de1e4150/128.), 128>>>(d_COUNT61b1de1c3110, COUNT61b1de1e4150);
uint64_t COUNT61b1de1c3110;
hipMemcpy(&COUNT61b1de1c3110, d_COUNT61b1de1c3110, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX61b1de1c3110;
hipMalloc(&d_MAT_IDX61b1de1c3110, sizeof(uint64_t));
hipMemset(d_MAT_IDX61b1de1c3110, 0, sizeof(uint64_t));
auto MAT61b1de1c3110date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT61b1de1c3110);
DBI32Type* d_MAT61b1de1c3110date__d_year;
hipMalloc(&d_MAT61b1de1c3110date__d_year, sizeof(DBI32Type) * COUNT61b1de1c3110);
auto MAT61b1de1c3110supplier__s_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT61b1de1c3110);
DBI16Type* d_MAT61b1de1c3110supplier__s_nation_encoded;
hipMalloc(&d_MAT61b1de1c3110supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT61b1de1c3110);
auto MAT61b1de1c3110part__p_category_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT61b1de1c3110);
DBI16Type* d_MAT61b1de1c3110part__p_category_encoded;
hipMalloc(&d_MAT61b1de1c3110part__p_category_encoded, sizeof(DBI16Type) * COUNT61b1de1c3110);
auto MAT61b1de1c3110aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT61b1de1c3110);
DBDecimalType* d_MAT61b1de1c3110aggr0__tmp_attr0;
hipMalloc(&d_MAT61b1de1c3110aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT61b1de1c3110);
main_61b1de2520c0<<<std::ceil((float)COUNT61b1de1e4150/128.), 128>>>(COUNT61b1de1e4150, d_MAT61b1de1c3110aggr0__tmp_attr0, d_MAT61b1de1c3110date__d_year, d_MAT61b1de1c3110part__p_category_encoded, d_MAT61b1de1c3110supplier__s_nation_encoded, d_MAT_IDX61b1de1c3110, d_aggr0__tmp_attr0, d_KEY_61b1de1e4150date__d_year, d_KEY_61b1de1e4150part__p_category_encoded, d_KEY_61b1de1e4150supplier__s_nation_encoded);
hipMemcpy(MAT61b1de1c3110date__d_year, d_MAT61b1de1c3110date__d_year, sizeof(DBI32Type) * COUNT61b1de1c3110, hipMemcpyDeviceToHost);
hipMemcpy(MAT61b1de1c3110supplier__s_nation_encoded, d_MAT61b1de1c3110supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT61b1de1c3110, hipMemcpyDeviceToHost);
hipMemcpy(MAT61b1de1c3110part__p_category_encoded, d_MAT61b1de1c3110part__p_category_encoded, sizeof(DBI16Type) * COUNT61b1de1c3110, hipMemcpyDeviceToHost);
hipMemcpy(MAT61b1de1c3110aggr0__tmp_attr0, d_MAT61b1de1c3110aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT61b1de1c3110, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT61b1de1c3110; i++) { std::cout << "" << MAT61b1de1c3110date__d_year[i];
std::cout << "," << supplier__s_nation_map[MAT61b1de1c3110supplier__s_nation_encoded[i]];
std::cout << "," << part__p_category_map[MAT61b1de1c3110part__p_category_encoded[i]];
std::cout << "," << MAT61b1de1c3110aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_61b1de22db60);
hipFree(d_BUF_IDX_61b1de22db60);
hipFree(d_COUNT61b1de22db60);
hipFree(d_BUF_61b1de22dc90);
hipFree(d_BUF_IDX_61b1de22dc90);
hipFree(d_COUNT61b1de22dc90);
hipFree(d_BUF_61b1de231120);
hipFree(d_BUF_IDX_61b1de231120);
hipFree(d_COUNT61b1de231120);
hipFree(d_BUF_61b1de20e760);
hipFree(d_BUF_IDX_61b1de20e760);
hipFree(d_COUNT61b1de20e760);
hipFree(d_KEY_61b1de1e4150date__d_year);
hipFree(d_KEY_61b1de1e4150part__p_category_encoded);
hipFree(d_KEY_61b1de1e4150supplier__s_nation_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT61b1de1c3110);
hipFree(d_MAT61b1de1c3110aggr0__tmp_attr0);
hipFree(d_MAT61b1de1c3110date__d_year);
hipFree(d_MAT61b1de1c3110part__p_category_encoded);
hipFree(d_MAT61b1de1c3110supplier__s_nation_encoded);
hipFree(d_MAT_IDX61b1de1c3110);
free(MAT61b1de1c3110aggr0__tmp_attr0);
free(MAT61b1de1c3110date__d_year);
free(MAT61b1de1c3110part__p_category_encoded);
free(MAT61b1de1c3110supplier__s_nation_encoded);
}