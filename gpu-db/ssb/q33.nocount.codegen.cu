#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBStringType* customer__c_city, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_city = customer__c_city[tid];
if (!((evaluatePredicate(reg_customer__c_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_customer__c_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0, buf_idx_0});
BUF_0[buf_idx_0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_INSERT HT_2, DBStringType* supplier__s_city, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_city = supplier__s_city[tid];
if (!((evaluatePredicate(reg_supplier__s_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_supplier__s_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_2 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_2 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_2 = atomicAdd((int*)BUF_IDX_2, 1);
HT_2.insert(cuco::pair{KEY_2, buf_idx_2});
BUF_2[buf_idx_2 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_0, uint64_t* BUF_2, uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_PROBE HT_0, HASHTABLE_PROBE HT_2, HASHTABLE_INSERT HT_4, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_0 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_0.for_each(KEY_0, [&] __device__ (auto const SLOT_0) {
auto const [slot_first0, slot_second0] = SLOT_0;
if (!(true)) return;
uint64_t KEY_2 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_2 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_2.for_each(KEY_2, [&] __device__ (auto const SLOT_2) {
auto const [slot_first2, slot_second2] = SLOT_2;
if (!(true)) return;
uint64_t KEY_4 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_4 |= reg_lineorder__lo_orderdate;
// Insert hash table kernel;
auto buf_idx_4 = atomicAdd((int*)BUF_IDX_4, 1);
HT_4.insert(cuco::pair{KEY_4, buf_idx_4});
BUF_4[buf_idx_4 * 3 + 0] = BUF_0[slot_second0 * 1 + 0];
BUF_4[buf_idx_4 * 3 + 1] = tid;
BUF_4[buf_idx_4 * 3 + 2] = BUF_2[slot_second2 * 1 + 0];
});
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_7(uint64_t* BUF_4, HASHTABLE_PROBE HT_4, HASHTABLE_FIND HT_6, DBI16Type* KEY_6customer__c_city_encoded, DBI32Type* KEY_6date__d_year, DBI16Type* KEY_6supplier__s_city_encoded, int* SLOT_COUNT_6, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size, DBDecimalType* lineorder__lo_revenue, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_4 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_4 |= reg_date__d_datekey;
//Probe Hash table
HT_4.for_each(KEY_4, [&] __device__ (auto const SLOT_4) {
auto const [slot_first4, slot_second4] = SLOT_4;
if (!(true)) return;
uint64_t KEY_6 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_4[slot_second4 * 3 + 0]];

KEY_6 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_4[slot_second4 * 3 + 2]];
KEY_6 <<= 16;
KEY_6 |= reg_supplier__s_city_encoded;
KEY_6 <<= 32;
KEY_6 |= reg_date__d_year;
//Aggregate in hashtable
auto buf_idx_6 = get_aggregation_slot(KEY_6, HT_6, SLOT_COUNT_6);
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_4[slot_second4 * 3 + 1]];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_6], reg_lineorder__lo_revenue);
KEY_6customer__c_city_encoded[buf_idx_6] = reg_customer__c_city_encoded;
KEY_6supplier__s_city_encoded[buf_idx_6] = reg_supplier__s_city_encoded;
KEY_6date__d_year[buf_idx_6] = reg_date__d_year;
});
}
__global__ void main_9(size_t COUNT6, DBDecimalType* MAT8aggr0__tmp_attr0, DBI16Type* MAT8customer__c_city_encoded, DBI32Type* MAT8date__d_year, DBI16Type* MAT8supplier__s_city_encoded, uint64_t* MAT_IDX8, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT6) return;
//Materialize buffers
auto mat_idx8 = atomicAdd((int*)MAT_IDX8, 1);
auto reg_customer__c_city_encoded = customer__c_city_encoded[tid];
MAT8customer__c_city_encoded[mat_idx8] = reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[tid];
MAT8supplier__s_city_encoded[mat_idx8] = reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[tid];
MAT8date__d_year[mat_idx8] = reg_date__d_year;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT8aggr0__tmp_attr0[mat_idx8] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
size_t COUNT0 = customer_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::experimental::static_multimap{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_customer__c_city, d_customer__c_custkey, customer_size);
hipFree(d_BUF_IDX_0);
size_t COUNT2 = supplier_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 1);
auto d_HT_2 = cuco::experimental::static_multimap{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_2, d_BUF_IDX_2, d_HT_2.ref(cuco::insert), d_supplier__s_city, d_supplier__s_suppkey, supplier_size);
hipFree(d_BUF_IDX_2);
size_t COUNT4 = lineorder_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 3);
auto d_HT_4 = cuco::experimental::static_multimap{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_0, d_BUF_2, d_BUF_4, d_BUF_IDX_4, d_HT_0.ref(cuco::for_each), d_HT_2.ref(cuco::for_each), d_HT_4.ref(cuco::insert), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size);
hipFree(d_BUF_IDX_4);
size_t COUNT6 = 132;
auto d_HT_6 = cuco::static_map{ (int)132*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_6;
hipMalloc(&d_SLOT_COUNT_6, sizeof(int));
hipMemset(d_SLOT_COUNT_6, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT6);
DBI16Type* d_KEY_6customer__c_city_encoded;
hipMalloc(&d_KEY_6customer__c_city_encoded, sizeof(DBI16Type) * COUNT6);
hipMemset(d_KEY_6customer__c_city_encoded, 0, sizeof(DBI16Type) * COUNT6);
DBI16Type* d_KEY_6supplier__s_city_encoded;
hipMalloc(&d_KEY_6supplier__s_city_encoded, sizeof(DBI16Type) * COUNT6);
hipMemset(d_KEY_6supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT6);
DBI32Type* d_KEY_6date__d_year;
hipMalloc(&d_KEY_6date__d_year, sizeof(DBI32Type) * COUNT6);
hipMemset(d_KEY_6date__d_year, 0, sizeof(DBI32Type) * COUNT6);
main_7<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_4, d_HT_4.ref(cuco::for_each), d_HT_6.ref(cuco::insert_and_find), d_KEY_6customer__c_city_encoded, d_KEY_6date__d_year, d_KEY_6supplier__s_city_encoded, d_SLOT_COUNT_6, d_aggr0__tmp_attr0, d_customer__c_city_encoded, d_date__d_datekey, d_date__d_year, date_size, d_lineorder__lo_revenue, d_supplier__s_city_encoded);
COUNT6 = d_HT_6.size();
size_t COUNT8 = COUNT6;
//Materialize buffers
uint64_t* d_MAT_IDX8;
hipMalloc(&d_MAT_IDX8, sizeof(uint64_t));
hipMemset(d_MAT_IDX8, 0, sizeof(uint64_t));
auto MAT8customer__c_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT8);
DBI16Type* d_MAT8customer__c_city_encoded;
hipMalloc(&d_MAT8customer__c_city_encoded, sizeof(DBI16Type) * COUNT8);
auto MAT8supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT8);
DBI16Type* d_MAT8supplier__s_city_encoded;
hipMalloc(&d_MAT8supplier__s_city_encoded, sizeof(DBI16Type) * COUNT8);
auto MAT8date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT8);
DBI32Type* d_MAT8date__d_year;
hipMalloc(&d_MAT8date__d_year, sizeof(DBI32Type) * COUNT8);
auto MAT8aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT8);
DBDecimalType* d_MAT8aggr0__tmp_attr0;
hipMalloc(&d_MAT8aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT8);
main_9<<<std::ceil((float)COUNT6/128.), 128>>>(COUNT6, d_MAT8aggr0__tmp_attr0, d_MAT8customer__c_city_encoded, d_MAT8date__d_year, d_MAT8supplier__s_city_encoded, d_MAT_IDX8, d_aggr0__tmp_attr0, d_KEY_6customer__c_city_encoded, d_KEY_6date__d_year, d_KEY_6supplier__s_city_encoded);
hipMemcpy(MAT8customer__c_city_encoded, d_MAT8customer__c_city_encoded, sizeof(DBI16Type) * COUNT8, hipMemcpyDeviceToHost);
hipMemcpy(MAT8supplier__s_city_encoded, d_MAT8supplier__s_city_encoded, sizeof(DBI16Type) * COUNT8, hipMemcpyDeviceToHost);
hipMemcpy(MAT8date__d_year, d_MAT8date__d_year, sizeof(DBI32Type) * COUNT8, hipMemcpyDeviceToHost);
hipMemcpy(MAT8aggr0__tmp_attr0, d_MAT8aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT8, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT8; i++) { std::cout << "" << customer__c_city_map[MAT8customer__c_city_encoded[i]];
std::cout << "," << supplier__s_city_map[MAT8supplier__s_city_encoded[i]];
std::cout << "," << MAT8date__d_year[i];
std::cout << "," << MAT8aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_KEY_6customer__c_city_encoded);
hipFree(d_KEY_6date__d_year);
hipFree(d_KEY_6supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_MAT8aggr0__tmp_attr0);
hipFree(d_MAT8customer__c_city_encoded);
hipFree(d_MAT8date__d_year);
hipFree(d_MAT8supplier__s_city_encoded);
hipFree(d_MAT_IDX8);
free(MAT8aggr0__tmp_attr0);
free(MAT8customer__c_city_encoded);
free(MAT8date__d_year);
free(MAT8supplier__s_city_encoded);
}