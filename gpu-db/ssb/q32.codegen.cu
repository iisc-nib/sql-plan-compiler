#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_616d1c136480(uint64_t* COUNT616d1c12c6b0, DBStringType* supplier__s_nation, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT616d1c12c6b0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_616d1c136480(uint64_t* BUF_616d1c12c6b0, uint64_t* BUF_IDX_616d1c12c6b0, HASHTABLE_INSERT HT_616d1c12c6b0, DBStringType* supplier__s_nation, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_616d1c12c6b0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_616d1c12c6b0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_616d1c12c6b0 = atomicAdd((int*)BUF_IDX_616d1c12c6b0, 1);
HT_616d1c12c6b0.insert(cuco::pair{KEY_616d1c12c6b0, buf_idx_616d1c12c6b0});
BUF_616d1c12c6b0[buf_idx_616d1c12c6b0 * 1 + 0] = tid;
}
__global__ void count_616d1c10f1a0(uint64_t* COUNT616d1c12d0d0, DBStringType* customer__c_nation, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_nation = customer__c_nation[tid];
if (!(evaluatePredicate(reg_customer__c_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT616d1c12d0d0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_616d1c10f1a0(uint64_t* BUF_616d1c12d0d0, uint64_t* BUF_IDX_616d1c12d0d0, HASHTABLE_INSERT HT_616d1c12d0d0, DBI32Type* customer__c_custkey, DBStringType* customer__c_nation, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_nation = customer__c_nation[tid];
if (!(evaluatePredicate(reg_customer__c_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_616d1c12d0d0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_616d1c12d0d0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_616d1c12d0d0 = atomicAdd((int*)BUF_IDX_616d1c12d0d0, 1);
HT_616d1c12d0d0.insert(cuco::pair{KEY_616d1c12d0d0, buf_idx_616d1c12d0d0});
BUF_616d1c12d0d0[buf_idx_616d1c12d0d0 * 1 + 0] = tid;
}
__global__ void count_616d1c13f370(uint64_t* COUNT616d1c12d190, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT616d1c12d190, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_616d1c13f370(uint64_t* BUF_616d1c12d190, uint64_t* BUF_IDX_616d1c12d190, HASHTABLE_INSERT HT_616d1c12d190, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_616d1c12d190 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_616d1c12d190 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_616d1c12d190 = atomicAdd((int*)BUF_IDX_616d1c12d190, 1);
HT_616d1c12d190.insert(cuco::pair{KEY_616d1c12d190, buf_idx_616d1c12d190});
BUF_616d1c12d190[buf_idx_616d1c12d190 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_616d1c10f740(uint64_t* BUF_616d1c12c6b0, uint64_t* BUF_616d1c12d0d0, uint64_t* BUF_616d1c12d190, HASHTABLE_INSERT HT_616d1c0df300, HASHTABLE_PROBE HT_616d1c12c6b0, HASHTABLE_PROBE HT_616d1c12d0d0, HASHTABLE_PROBE HT_616d1c12d190, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_616d1c12c6b0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_616d1c12c6b0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_616d1c12c6b0.for_each(KEY_616d1c12c6b0, [&] __device__ (auto const SLOT_616d1c12c6b0) {

auto const [slot_first616d1c12c6b0, slot_second616d1c12c6b0] = SLOT_616d1c12c6b0;
if (!(true)) return;
uint64_t KEY_616d1c12d0d0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_616d1c12d0d0 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_616d1c12d0d0.for_each(KEY_616d1c12d0d0, [&] __device__ (auto const SLOT_616d1c12d0d0) {

auto const [slot_first616d1c12d0d0, slot_second616d1c12d0d0] = SLOT_616d1c12d0d0;
if (!(true)) return;
uint64_t KEY_616d1c12d190 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_616d1c12d190 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_616d1c12d190.for_each(KEY_616d1c12d190, [&] __device__ (auto const SLOT_616d1c12d190) {

auto const [slot_first616d1c12d190, slot_second616d1c12d190] = SLOT_616d1c12d190;
if (!(true)) return;
uint64_t KEY_616d1c0df300 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_616d1c12d0d0[slot_second616d1c12d0d0 * 1 + 0]];

KEY_616d1c0df300 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_616d1c12c6b0[slot_second616d1c12c6b0 * 1 + 0]];
KEY_616d1c0df300 <<= 16;
KEY_616d1c0df300 |= reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[BUF_616d1c12d190[slot_second616d1c12d190 * 1 + 0]];
KEY_616d1c0df300 <<= 32;
KEY_616d1c0df300 |= reg_date__d_year;
//Create aggregation hash table
HT_616d1c0df300.insert(cuco::pair{KEY_616d1c0df300, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_616d1c10f740(uint64_t* BUF_616d1c12c6b0, uint64_t* BUF_616d1c12d0d0, uint64_t* BUF_616d1c12d190, HASHTABLE_FIND HT_616d1c0df300, HASHTABLE_PROBE HT_616d1c12c6b0, HASHTABLE_PROBE HT_616d1c12d0d0, HASHTABLE_PROBE HT_616d1c12d190, DBI16Type* KEY_616d1c0df300customer__c_city_encoded, DBI32Type* KEY_616d1c0df300date__d_year, DBI16Type* KEY_616d1c0df300supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_616d1c12c6b0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_616d1c12c6b0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_616d1c12c6b0.for_each(KEY_616d1c12c6b0, [&] __device__ (auto const SLOT_616d1c12c6b0) {
auto const [slot_first616d1c12c6b0, slot_second616d1c12c6b0] = SLOT_616d1c12c6b0;
if (!(true)) return;
uint64_t KEY_616d1c12d0d0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_616d1c12d0d0 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_616d1c12d0d0.for_each(KEY_616d1c12d0d0, [&] __device__ (auto const SLOT_616d1c12d0d0) {
auto const [slot_first616d1c12d0d0, slot_second616d1c12d0d0] = SLOT_616d1c12d0d0;
if (!(true)) return;
uint64_t KEY_616d1c12d190 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_616d1c12d190 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_616d1c12d190.for_each(KEY_616d1c12d190, [&] __device__ (auto const SLOT_616d1c12d190) {
auto const [slot_first616d1c12d190, slot_second616d1c12d190] = SLOT_616d1c12d190;
if (!(true)) return;
uint64_t KEY_616d1c0df300 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_616d1c12d0d0[slot_second616d1c12d0d0 * 1 + 0]];

KEY_616d1c0df300 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_616d1c12c6b0[slot_second616d1c12c6b0 * 1 + 0]];
KEY_616d1c0df300 <<= 16;
KEY_616d1c0df300 |= reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[BUF_616d1c12d190[slot_second616d1c12d190 * 1 + 0]];
KEY_616d1c0df300 <<= 32;
KEY_616d1c0df300 |= reg_date__d_year;
//Aggregate in hashtable
auto buf_idx_616d1c0df300 = HT_616d1c0df300.find(KEY_616d1c0df300)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_616d1c0df300], reg_lineorder__lo_revenue);
KEY_616d1c0df300customer__c_city_encoded[buf_idx_616d1c0df300] = reg_customer__c_city_encoded;
KEY_616d1c0df300supplier__s_city_encoded[buf_idx_616d1c0df300] = reg_supplier__s_city_encoded;
KEY_616d1c0df300date__d_year[buf_idx_616d1c0df300] = reg_date__d_year;
});
});
});
}
__global__ void count_616d1c149660(uint64_t* COUNT616d1c0bac60, size_t COUNT616d1c0df300) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT616d1c0df300) return;
//Materialize count
atomicAdd((int*)COUNT616d1c0bac60, 1);
}
__global__ void main_616d1c149660(size_t COUNT616d1c0df300, DBDecimalType* MAT616d1c0bac60aggr0__tmp_attr0, DBI16Type* MAT616d1c0bac60customer__c_city_encoded, DBI32Type* MAT616d1c0bac60date__d_year, DBI16Type* MAT616d1c0bac60supplier__s_city_encoded, uint64_t* MAT_IDX616d1c0bac60, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT616d1c0df300) return;
//Materialize buffers
auto mat_idx616d1c0bac60 = atomicAdd((int*)MAT_IDX616d1c0bac60, 1);
auto reg_customer__c_city_encoded = customer__c_city_encoded[tid];
MAT616d1c0bac60customer__c_city_encoded[mat_idx616d1c0bac60] = reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[tid];
MAT616d1c0bac60supplier__s_city_encoded[mat_idx616d1c0bac60] = reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[tid];
MAT616d1c0bac60date__d_year[mat_idx616d1c0bac60] = reg_date__d_year;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT616d1c0bac60aggr0__tmp_attr0[mat_idx616d1c0bac60] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT616d1c12c6b0;
hipMalloc(&d_COUNT616d1c12c6b0, sizeof(uint64_t));
hipMemset(d_COUNT616d1c12c6b0, 0, sizeof(uint64_t));
count_616d1c136480<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT616d1c12c6b0, d_supplier__s_nation, supplier_size);
uint64_t COUNT616d1c12c6b0;
hipMemcpy(&COUNT616d1c12c6b0, d_COUNT616d1c12c6b0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_616d1c12c6b0;
hipMalloc(&d_BUF_IDX_616d1c12c6b0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_616d1c12c6b0, 0, sizeof(uint64_t));
uint64_t* d_BUF_616d1c12c6b0;
hipMalloc(&d_BUF_616d1c12c6b0, sizeof(uint64_t) * COUNT616d1c12c6b0 * 1);
auto d_HT_616d1c12c6b0 = cuco::experimental::static_multimap{ (int)COUNT616d1c12c6b0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_616d1c136480<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_616d1c12c6b0, d_BUF_IDX_616d1c12c6b0, d_HT_616d1c12c6b0.ref(cuco::insert), d_supplier__s_nation, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT616d1c12d0d0;
hipMalloc(&d_COUNT616d1c12d0d0, sizeof(uint64_t));
hipMemset(d_COUNT616d1c12d0d0, 0, sizeof(uint64_t));
count_616d1c10f1a0<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT616d1c12d0d0, d_customer__c_nation, customer_size);
uint64_t COUNT616d1c12d0d0;
hipMemcpy(&COUNT616d1c12d0d0, d_COUNT616d1c12d0d0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_616d1c12d0d0;
hipMalloc(&d_BUF_IDX_616d1c12d0d0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_616d1c12d0d0, 0, sizeof(uint64_t));
uint64_t* d_BUF_616d1c12d0d0;
hipMalloc(&d_BUF_616d1c12d0d0, sizeof(uint64_t) * COUNT616d1c12d0d0 * 1);
auto d_HT_616d1c12d0d0 = cuco::experimental::static_multimap{ (int)COUNT616d1c12d0d0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_616d1c10f1a0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_616d1c12d0d0, d_BUF_IDX_616d1c12d0d0, d_HT_616d1c12d0d0.ref(cuco::insert), d_customer__c_custkey, d_customer__c_nation, customer_size);
//Materialize count
uint64_t* d_COUNT616d1c12d190;
hipMalloc(&d_COUNT616d1c12d190, sizeof(uint64_t));
hipMemset(d_COUNT616d1c12d190, 0, sizeof(uint64_t));
count_616d1c13f370<<<std::ceil((float)date_size/32.), 32>>>(d_COUNT616d1c12d190, d_date__d_year, date_size);
uint64_t COUNT616d1c12d190;
hipMemcpy(&COUNT616d1c12d190, d_COUNT616d1c12d190, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_616d1c12d190;
hipMalloc(&d_BUF_IDX_616d1c12d190, sizeof(uint64_t));
hipMemset(d_BUF_IDX_616d1c12d190, 0, sizeof(uint64_t));
uint64_t* d_BUF_616d1c12d190;
hipMalloc(&d_BUF_616d1c12d190, sizeof(uint64_t) * COUNT616d1c12d190 * 1);
auto d_HT_616d1c12d190 = cuco::experimental::static_multimap{ (int)COUNT616d1c12d190*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_616d1c13f370<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_616d1c12d190, d_BUF_IDX_616d1c12d190, d_HT_616d1c12d190.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Create aggregation hash table
auto d_HT_616d1c0df300 = cuco::static_map{ (int)5679*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_616d1c10f740<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_616d1c12c6b0, d_BUF_616d1c12d0d0, d_BUF_616d1c12d190, d_HT_616d1c0df300.ref(cuco::insert), d_HT_616d1c12c6b0.ref(cuco::for_each), d_HT_616d1c12d0d0.ref(cuco::for_each), d_HT_616d1c12d190.ref(cuco::for_each), d_customer__c_city_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
size_t COUNT616d1c0df300 = d_HT_616d1c0df300.size();
thrust::device_vector<int64_t> keys_616d1c0df300(COUNT616d1c0df300), vals_616d1c0df300(COUNT616d1c0df300);
d_HT_616d1c0df300.retrieve_all(keys_616d1c0df300.begin(), vals_616d1c0df300.begin());
d_HT_616d1c0df300.clear();
int64_t* raw_keys616d1c0df300 = thrust::raw_pointer_cast(keys_616d1c0df300.data());
insertKeys<<<std::ceil((float)COUNT616d1c0df300/32.), 32>>>(raw_keys616d1c0df300, d_HT_616d1c0df300.ref(cuco::insert), COUNT616d1c0df300);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT616d1c0df300);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT616d1c0df300);
DBI16Type* d_KEY_616d1c0df300customer__c_city_encoded;
hipMalloc(&d_KEY_616d1c0df300customer__c_city_encoded, sizeof(DBI16Type) * COUNT616d1c0df300);
hipMemset(d_KEY_616d1c0df300customer__c_city_encoded, 0, sizeof(DBI16Type) * COUNT616d1c0df300);
DBI16Type* d_KEY_616d1c0df300supplier__s_city_encoded;
hipMalloc(&d_KEY_616d1c0df300supplier__s_city_encoded, sizeof(DBI16Type) * COUNT616d1c0df300);
hipMemset(d_KEY_616d1c0df300supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT616d1c0df300);
DBI32Type* d_KEY_616d1c0df300date__d_year;
hipMalloc(&d_KEY_616d1c0df300date__d_year, sizeof(DBI32Type) * COUNT616d1c0df300);
hipMemset(d_KEY_616d1c0df300date__d_year, 0, sizeof(DBI32Type) * COUNT616d1c0df300);
main_616d1c10f740<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_616d1c12c6b0, d_BUF_616d1c12d0d0, d_BUF_616d1c12d190, d_HT_616d1c0df300.ref(cuco::find), d_HT_616d1c12c6b0.ref(cuco::for_each), d_HT_616d1c12d0d0.ref(cuco::for_each), d_HT_616d1c12d190.ref(cuco::for_each), d_KEY_616d1c0df300customer__c_city_encoded, d_KEY_616d1c0df300date__d_year, d_KEY_616d1c0df300supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_city_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
//Materialize count
uint64_t* d_COUNT616d1c0bac60;
hipMalloc(&d_COUNT616d1c0bac60, sizeof(uint64_t));
hipMemset(d_COUNT616d1c0bac60, 0, sizeof(uint64_t));
count_616d1c149660<<<std::ceil((float)COUNT616d1c0df300/32.), 32>>>(d_COUNT616d1c0bac60, COUNT616d1c0df300);
uint64_t COUNT616d1c0bac60;
hipMemcpy(&COUNT616d1c0bac60, d_COUNT616d1c0bac60, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX616d1c0bac60;
hipMalloc(&d_MAT_IDX616d1c0bac60, sizeof(uint64_t));
hipMemset(d_MAT_IDX616d1c0bac60, 0, sizeof(uint64_t));
auto MAT616d1c0bac60customer__c_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT616d1c0bac60);
DBI16Type* d_MAT616d1c0bac60customer__c_city_encoded;
hipMalloc(&d_MAT616d1c0bac60customer__c_city_encoded, sizeof(DBI16Type) * COUNT616d1c0bac60);
auto MAT616d1c0bac60supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT616d1c0bac60);
DBI16Type* d_MAT616d1c0bac60supplier__s_city_encoded;
hipMalloc(&d_MAT616d1c0bac60supplier__s_city_encoded, sizeof(DBI16Type) * COUNT616d1c0bac60);
auto MAT616d1c0bac60date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT616d1c0bac60);
DBI32Type* d_MAT616d1c0bac60date__d_year;
hipMalloc(&d_MAT616d1c0bac60date__d_year, sizeof(DBI32Type) * COUNT616d1c0bac60);
auto MAT616d1c0bac60aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT616d1c0bac60);
DBDecimalType* d_MAT616d1c0bac60aggr0__tmp_attr0;
hipMalloc(&d_MAT616d1c0bac60aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT616d1c0bac60);
main_616d1c149660<<<std::ceil((float)COUNT616d1c0df300/32.), 32>>>(COUNT616d1c0df300, d_MAT616d1c0bac60aggr0__tmp_attr0, d_MAT616d1c0bac60customer__c_city_encoded, d_MAT616d1c0bac60date__d_year, d_MAT616d1c0bac60supplier__s_city_encoded, d_MAT_IDX616d1c0bac60, d_aggr0__tmp_attr0, d_KEY_616d1c0df300customer__c_city_encoded, d_KEY_616d1c0df300date__d_year, d_KEY_616d1c0df300supplier__s_city_encoded);
hipMemcpy(MAT616d1c0bac60customer__c_city_encoded, d_MAT616d1c0bac60customer__c_city_encoded, sizeof(DBI16Type) * COUNT616d1c0bac60, hipMemcpyDeviceToHost);
hipMemcpy(MAT616d1c0bac60supplier__s_city_encoded, d_MAT616d1c0bac60supplier__s_city_encoded, sizeof(DBI16Type) * COUNT616d1c0bac60, hipMemcpyDeviceToHost);
hipMemcpy(MAT616d1c0bac60date__d_year, d_MAT616d1c0bac60date__d_year, sizeof(DBI32Type) * COUNT616d1c0bac60, hipMemcpyDeviceToHost);
hipMemcpy(MAT616d1c0bac60aggr0__tmp_attr0, d_MAT616d1c0bac60aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT616d1c0bac60, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT616d1c0bac60; i++) { std::cout << customer__c_city_map[MAT616d1c0bac60customer__c_city_encoded[i]] << "\t";
std::cout << supplier__s_city_map[MAT616d1c0bac60supplier__s_city_encoded[i]] << "\t";
std::cout << MAT616d1c0bac60date__d_year[i] << "\t";
std::cout << MAT616d1c0bac60aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_616d1c12c6b0);
hipFree(d_BUF_IDX_616d1c12c6b0);
hipFree(d_COUNT616d1c12c6b0);
hipFree(d_BUF_616d1c12d0d0);
hipFree(d_BUF_IDX_616d1c12d0d0);
hipFree(d_COUNT616d1c12d0d0);
hipFree(d_BUF_616d1c12d190);
hipFree(d_BUF_IDX_616d1c12d190);
hipFree(d_COUNT616d1c12d190);
hipFree(d_KEY_616d1c0df300customer__c_city_encoded);
hipFree(d_KEY_616d1c0df300date__d_year);
hipFree(d_KEY_616d1c0df300supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT616d1c0bac60);
hipFree(d_MAT616d1c0bac60aggr0__tmp_attr0);
hipFree(d_MAT616d1c0bac60customer__c_city_encoded);
hipFree(d_MAT616d1c0bac60date__d_year);
hipFree(d_MAT616d1c0bac60supplier__s_city_encoded);
hipFree(d_MAT_IDX616d1c0bac60);
free(MAT616d1c0bac60aggr0__tmp_attr0);
free(MAT616d1c0bac60customer__c_city_encoded);
free(MAT616d1c0bac60date__d_year);
free(MAT616d1c0bac60supplier__s_city_encoded);
}