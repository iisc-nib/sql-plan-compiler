#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_628a78872b00(uint64_t* COUNT628a78867080, DBStringType* supplier__s_nation, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT628a78867080, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_628a78872b00(uint64_t* BUF_628a78867080, uint64_t* BUF_IDX_628a78867080, HASHTABLE_INSERT HT_628a78867080, DBStringType* supplier__s_nation, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_628a78867080 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_628a78867080 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_628a78867080 = atomicAdd((int*)BUF_IDX_628a78867080, 1);
HT_628a78867080.insert(cuco::pair{KEY_628a78867080, buf_idx_628a78867080});
BUF_628a78867080[buf_idx_628a78867080 * 1 + 0] = tid;
}
__global__ void count_628a7884a9c0(uint64_t* COUNT628a788691f0, DBStringType* customer__c_nation, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_nation = customer__c_nation[tid];
if (!(evaluatePredicate(reg_customer__c_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT628a788691f0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_628a7884a9c0(uint64_t* BUF_628a788691f0, uint64_t* BUF_IDX_628a788691f0, HASHTABLE_INSERT HT_628a788691f0, DBI32Type* customer__c_custkey, DBStringType* customer__c_nation, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_nation = customer__c_nation[tid];
if (!(evaluatePredicate(reg_customer__c_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_628a788691f0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_628a788691f0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_628a788691f0 = atomicAdd((int*)BUF_IDX_628a788691f0, 1);
HT_628a788691f0.insert(cuco::pair{KEY_628a788691f0, buf_idx_628a788691f0});
BUF_628a788691f0[buf_idx_628a788691f0 * 1 + 0] = tid;
}
__global__ void count_628a7887b9f0(uint64_t* COUNT628a78866990, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT628a78866990, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_628a7887b9f0(uint64_t* BUF_628a78866990, uint64_t* BUF_IDX_628a78866990, HASHTABLE_INSERT HT_628a78866990, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_628a78866990 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_628a78866990 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_628a78866990 = atomicAdd((int*)BUF_IDX_628a78866990, 1);
HT_628a78866990.insert(cuco::pair{KEY_628a78866990, buf_idx_628a78866990});
BUF_628a78866990[buf_idx_628a78866990 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_628a7884af60(uint64_t* BUF_628a78866990, uint64_t* BUF_628a78867080, uint64_t* BUF_628a788691f0, HASHTABLE_INSERT HT_628a7881a2f0, HASHTABLE_PROBE HT_628a78866990, HASHTABLE_PROBE HT_628a78867080, HASHTABLE_PROBE HT_628a788691f0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_628a78867080 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_628a78867080 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_628a78867080.for_each(KEY_628a78867080, [&] __device__ (auto const SLOT_628a78867080) {

auto const [slot_first628a78867080, slot_second628a78867080] = SLOT_628a78867080;
if (!(true)) return;
uint64_t KEY_628a788691f0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_628a788691f0 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_628a788691f0.for_each(KEY_628a788691f0, [&] __device__ (auto const SLOT_628a788691f0) {

auto const [slot_first628a788691f0, slot_second628a788691f0] = SLOT_628a788691f0;
if (!(true)) return;
uint64_t KEY_628a78866990 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_628a78866990 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_628a78866990.for_each(KEY_628a78866990, [&] __device__ (auto const SLOT_628a78866990) {

auto const [slot_first628a78866990, slot_second628a78866990] = SLOT_628a78866990;
if (!(true)) return;
uint64_t KEY_628a7881a2f0 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_628a788691f0[slot_second628a788691f0 * 1 + 0]];

KEY_628a7881a2f0 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_628a78867080[slot_second628a78867080 * 1 + 0]];
KEY_628a7881a2f0 <<= 16;
KEY_628a7881a2f0 |= reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[BUF_628a78866990[slot_second628a78866990 * 1 + 0]];
KEY_628a7881a2f0 <<= 32;
KEY_628a7881a2f0 |= reg_date__d_year;
//Create aggregation hash table
HT_628a7881a2f0.insert(cuco::pair{KEY_628a7881a2f0, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_628a7884af60(uint64_t* BUF_628a78866990, uint64_t* BUF_628a78867080, uint64_t* BUF_628a788691f0, HASHTABLE_FIND HT_628a7881a2f0, HASHTABLE_PROBE HT_628a78866990, HASHTABLE_PROBE HT_628a78867080, HASHTABLE_PROBE HT_628a788691f0, DBI16Type* KEY_628a7881a2f0customer__c_city_encoded, DBI32Type* KEY_628a7881a2f0date__d_year, DBI16Type* KEY_628a7881a2f0supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_628a78867080 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_628a78867080 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_628a78867080.for_each(KEY_628a78867080, [&] __device__ (auto const SLOT_628a78867080) {
auto const [slot_first628a78867080, slot_second628a78867080] = SLOT_628a78867080;
if (!(true)) return;
uint64_t KEY_628a788691f0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_628a788691f0 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_628a788691f0.for_each(KEY_628a788691f0, [&] __device__ (auto const SLOT_628a788691f0) {
auto const [slot_first628a788691f0, slot_second628a788691f0] = SLOT_628a788691f0;
if (!(true)) return;
uint64_t KEY_628a78866990 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_628a78866990 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_628a78866990.for_each(KEY_628a78866990, [&] __device__ (auto const SLOT_628a78866990) {
auto const [slot_first628a78866990, slot_second628a78866990] = SLOT_628a78866990;
if (!(true)) return;
uint64_t KEY_628a7881a2f0 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_628a788691f0[slot_second628a788691f0 * 1 + 0]];

KEY_628a7881a2f0 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_628a78867080[slot_second628a78867080 * 1 + 0]];
KEY_628a7881a2f0 <<= 16;
KEY_628a7881a2f0 |= reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[BUF_628a78866990[slot_second628a78866990 * 1 + 0]];
KEY_628a7881a2f0 <<= 32;
KEY_628a7881a2f0 |= reg_date__d_year;
//Aggregate in hashtable
auto buf_idx_628a7881a2f0 = HT_628a7881a2f0.find(KEY_628a7881a2f0)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_628a7881a2f0], reg_lineorder__lo_revenue);
KEY_628a7881a2f0customer__c_city_encoded[buf_idx_628a7881a2f0] = reg_customer__c_city_encoded;
KEY_628a7881a2f0supplier__s_city_encoded[buf_idx_628a7881a2f0] = reg_supplier__s_city_encoded;
KEY_628a7881a2f0date__d_year[buf_idx_628a7881a2f0] = reg_date__d_year;
});
});
});
}
__global__ void count_628a78885eb0(uint64_t* COUNT628a787f6920, size_t COUNT628a7881a2f0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT628a7881a2f0) return;
//Materialize count
atomicAdd((int*)COUNT628a787f6920, 1);
}
__global__ void main_628a78885eb0(size_t COUNT628a7881a2f0, DBDecimalType* MAT628a787f6920aggr0__tmp_attr0, DBI16Type* MAT628a787f6920customer__c_city_encoded, DBI32Type* MAT628a787f6920date__d_year, DBI16Type* MAT628a787f6920supplier__s_city_encoded, uint64_t* MAT_IDX628a787f6920, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT628a7881a2f0) return;
//Materialize buffers
auto mat_idx628a787f6920 = atomicAdd((int*)MAT_IDX628a787f6920, 1);
auto reg_customer__c_city_encoded = customer__c_city_encoded[tid];
MAT628a787f6920customer__c_city_encoded[mat_idx628a787f6920] = reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[tid];
MAT628a787f6920supplier__s_city_encoded[mat_idx628a787f6920] = reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[tid];
MAT628a787f6920date__d_year[mat_idx628a787f6920] = reg_date__d_year;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT628a787f6920aggr0__tmp_attr0[mat_idx628a787f6920] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT628a78867080;
hipMalloc(&d_COUNT628a78867080, sizeof(uint64_t));
hipMemset(d_COUNT628a78867080, 0, sizeof(uint64_t));
count_628a78872b00<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT628a78867080, d_supplier__s_nation, supplier_size);
uint64_t COUNT628a78867080;
hipMemcpy(&COUNT628a78867080, d_COUNT628a78867080, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_628a78867080;
hipMalloc(&d_BUF_IDX_628a78867080, sizeof(uint64_t));
hipMemset(d_BUF_IDX_628a78867080, 0, sizeof(uint64_t));
uint64_t* d_BUF_628a78867080;
hipMalloc(&d_BUF_628a78867080, sizeof(uint64_t) * COUNT628a78867080 * 1);
auto d_HT_628a78867080 = cuco::experimental::static_multimap{ (int)COUNT628a78867080*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_628a78872b00<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_628a78867080, d_BUF_IDX_628a78867080, d_HT_628a78867080.ref(cuco::insert), d_supplier__s_nation, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT628a788691f0;
hipMalloc(&d_COUNT628a788691f0, sizeof(uint64_t));
hipMemset(d_COUNT628a788691f0, 0, sizeof(uint64_t));
count_628a7884a9c0<<<std::ceil((float)customer_size/128.), 128>>>(d_COUNT628a788691f0, d_customer__c_nation, customer_size);
uint64_t COUNT628a788691f0;
hipMemcpy(&COUNT628a788691f0, d_COUNT628a788691f0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_628a788691f0;
hipMalloc(&d_BUF_IDX_628a788691f0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_628a788691f0, 0, sizeof(uint64_t));
uint64_t* d_BUF_628a788691f0;
hipMalloc(&d_BUF_628a788691f0, sizeof(uint64_t) * COUNT628a788691f0 * 1);
auto d_HT_628a788691f0 = cuco::experimental::static_multimap{ (int)COUNT628a788691f0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_628a7884a9c0<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_628a788691f0, d_BUF_IDX_628a788691f0, d_HT_628a788691f0.ref(cuco::insert), d_customer__c_custkey, d_customer__c_nation, customer_size);
//Materialize count
uint64_t* d_COUNT628a78866990;
hipMalloc(&d_COUNT628a78866990, sizeof(uint64_t));
hipMemset(d_COUNT628a78866990, 0, sizeof(uint64_t));
count_628a7887b9f0<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT628a78866990, d_date__d_year, date_size);
uint64_t COUNT628a78866990;
hipMemcpy(&COUNT628a78866990, d_COUNT628a78866990, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_628a78866990;
hipMalloc(&d_BUF_IDX_628a78866990, sizeof(uint64_t));
hipMemset(d_BUF_IDX_628a78866990, 0, sizeof(uint64_t));
uint64_t* d_BUF_628a78866990;
hipMalloc(&d_BUF_628a78866990, sizeof(uint64_t) * COUNT628a78866990 * 1);
auto d_HT_628a78866990 = cuco::experimental::static_multimap{ (int)COUNT628a78866990*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_628a7887b9f0<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_628a78866990, d_BUF_IDX_628a78866990, d_HT_628a78866990.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Create aggregation hash table
auto d_HT_628a7881a2f0 = cuco::static_map{ (int)5679*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_628a7884af60<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_628a78866990, d_BUF_628a78867080, d_BUF_628a788691f0, d_HT_628a7881a2f0.ref(cuco::insert), d_HT_628a78866990.ref(cuco::for_each), d_HT_628a78867080.ref(cuco::for_each), d_HT_628a788691f0.ref(cuco::for_each), d_customer__c_city_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
size_t COUNT628a7881a2f0 = d_HT_628a7881a2f0.size();
thrust::device_vector<int64_t> keys_628a7881a2f0(COUNT628a7881a2f0), vals_628a7881a2f0(COUNT628a7881a2f0);
d_HT_628a7881a2f0.retrieve_all(keys_628a7881a2f0.begin(), vals_628a7881a2f0.begin());
d_HT_628a7881a2f0.clear();
int64_t* raw_keys628a7881a2f0 = thrust::raw_pointer_cast(keys_628a7881a2f0.data());
insertKeys<<<std::ceil((float)COUNT628a7881a2f0/128.), 128>>>(raw_keys628a7881a2f0, d_HT_628a7881a2f0.ref(cuco::insert), COUNT628a7881a2f0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT628a7881a2f0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT628a7881a2f0);
DBI16Type* d_KEY_628a7881a2f0customer__c_city_encoded;
hipMalloc(&d_KEY_628a7881a2f0customer__c_city_encoded, sizeof(DBI16Type) * COUNT628a7881a2f0);
hipMemset(d_KEY_628a7881a2f0customer__c_city_encoded, 0, sizeof(DBI16Type) * COUNT628a7881a2f0);
DBI16Type* d_KEY_628a7881a2f0supplier__s_city_encoded;
hipMalloc(&d_KEY_628a7881a2f0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT628a7881a2f0);
hipMemset(d_KEY_628a7881a2f0supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT628a7881a2f0);
DBI32Type* d_KEY_628a7881a2f0date__d_year;
hipMalloc(&d_KEY_628a7881a2f0date__d_year, sizeof(DBI32Type) * COUNT628a7881a2f0);
hipMemset(d_KEY_628a7881a2f0date__d_year, 0, sizeof(DBI32Type) * COUNT628a7881a2f0);
main_628a7884af60<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_628a78866990, d_BUF_628a78867080, d_BUF_628a788691f0, d_HT_628a7881a2f0.ref(cuco::find), d_HT_628a78866990.ref(cuco::for_each), d_HT_628a78867080.ref(cuco::for_each), d_HT_628a788691f0.ref(cuco::for_each), d_KEY_628a7881a2f0customer__c_city_encoded, d_KEY_628a7881a2f0date__d_year, d_KEY_628a7881a2f0supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_city_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
//Materialize count
uint64_t* d_COUNT628a787f6920;
hipMalloc(&d_COUNT628a787f6920, sizeof(uint64_t));
hipMemset(d_COUNT628a787f6920, 0, sizeof(uint64_t));
count_628a78885eb0<<<std::ceil((float)COUNT628a7881a2f0/128.), 128>>>(d_COUNT628a787f6920, COUNT628a7881a2f0);
uint64_t COUNT628a787f6920;
hipMemcpy(&COUNT628a787f6920, d_COUNT628a787f6920, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX628a787f6920;
hipMalloc(&d_MAT_IDX628a787f6920, sizeof(uint64_t));
hipMemset(d_MAT_IDX628a787f6920, 0, sizeof(uint64_t));
auto MAT628a787f6920customer__c_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT628a787f6920);
DBI16Type* d_MAT628a787f6920customer__c_city_encoded;
hipMalloc(&d_MAT628a787f6920customer__c_city_encoded, sizeof(DBI16Type) * COUNT628a787f6920);
auto MAT628a787f6920supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT628a787f6920);
DBI16Type* d_MAT628a787f6920supplier__s_city_encoded;
hipMalloc(&d_MAT628a787f6920supplier__s_city_encoded, sizeof(DBI16Type) * COUNT628a787f6920);
auto MAT628a787f6920date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT628a787f6920);
DBI32Type* d_MAT628a787f6920date__d_year;
hipMalloc(&d_MAT628a787f6920date__d_year, sizeof(DBI32Type) * COUNT628a787f6920);
auto MAT628a787f6920aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT628a787f6920);
DBDecimalType* d_MAT628a787f6920aggr0__tmp_attr0;
hipMalloc(&d_MAT628a787f6920aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT628a787f6920);
main_628a78885eb0<<<std::ceil((float)COUNT628a7881a2f0/128.), 128>>>(COUNT628a7881a2f0, d_MAT628a787f6920aggr0__tmp_attr0, d_MAT628a787f6920customer__c_city_encoded, d_MAT628a787f6920date__d_year, d_MAT628a787f6920supplier__s_city_encoded, d_MAT_IDX628a787f6920, d_aggr0__tmp_attr0, d_KEY_628a7881a2f0customer__c_city_encoded, d_KEY_628a7881a2f0date__d_year, d_KEY_628a7881a2f0supplier__s_city_encoded);
hipMemcpy(MAT628a787f6920customer__c_city_encoded, d_MAT628a787f6920customer__c_city_encoded, sizeof(DBI16Type) * COUNT628a787f6920, hipMemcpyDeviceToHost);
hipMemcpy(MAT628a787f6920supplier__s_city_encoded, d_MAT628a787f6920supplier__s_city_encoded, sizeof(DBI16Type) * COUNT628a787f6920, hipMemcpyDeviceToHost);
hipMemcpy(MAT628a787f6920date__d_year, d_MAT628a787f6920date__d_year, sizeof(DBI32Type) * COUNT628a787f6920, hipMemcpyDeviceToHost);
hipMemcpy(MAT628a787f6920aggr0__tmp_attr0, d_MAT628a787f6920aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT628a787f6920, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT628a787f6920; i++) { std::cout << "" << customer__c_city_map[MAT628a787f6920customer__c_city_encoded[i]];
std::cout << "," << supplier__s_city_map[MAT628a787f6920supplier__s_city_encoded[i]];
std::cout << "," << MAT628a787f6920date__d_year[i];
std::cout << "," << MAT628a787f6920aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_628a78867080);
hipFree(d_BUF_IDX_628a78867080);
hipFree(d_COUNT628a78867080);
hipFree(d_BUF_628a788691f0);
hipFree(d_BUF_IDX_628a788691f0);
hipFree(d_COUNT628a788691f0);
hipFree(d_BUF_628a78866990);
hipFree(d_BUF_IDX_628a78866990);
hipFree(d_COUNT628a78866990);
hipFree(d_KEY_628a7881a2f0customer__c_city_encoded);
hipFree(d_KEY_628a7881a2f0date__d_year);
hipFree(d_KEY_628a7881a2f0supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT628a787f6920);
hipFree(d_MAT628a787f6920aggr0__tmp_attr0);
hipFree(d_MAT628a787f6920customer__c_city_encoded);
hipFree(d_MAT628a787f6920date__d_year);
hipFree(d_MAT628a787f6920supplier__s_city_encoded);
hipFree(d_MAT_IDX628a787f6920);
free(MAT628a787f6920aggr0__tmp_attr0);
free(MAT628a787f6920customer__c_city_encoded);
free(MAT628a787f6920date__d_year);
free(MAT628a787f6920supplier__s_city_encoded);
}