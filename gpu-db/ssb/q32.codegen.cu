#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_64f6e8a78790(uint64_t* COUNT64f6e8a6cf30, DBStringType* supplier__s_nation, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT64f6e8a6cf30, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_64f6e8a78790(uint64_t* BUF_64f6e8a6cf30, uint64_t* BUF_IDX_64f6e8a6cf30, HASHTABLE_INSERT HT_64f6e8a6cf30, DBStringType* supplier__s_nation, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_64f6e8a6cf30 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_64f6e8a6cf30 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_64f6e8a6cf30 = atomicAdd((int*)BUF_IDX_64f6e8a6cf30, 1);
HT_64f6e8a6cf30.insert(cuco::pair{KEY_64f6e8a6cf30, buf_idx_64f6e8a6cf30});
BUF_64f6e8a6cf30[buf_idx_64f6e8a6cf30 * 1 + 0] = tid;
}
__global__ void count_64f6e8a516b0(uint64_t* COUNT64f6e8a6f700, DBStringType* customer__c_nation, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_nation = customer__c_nation[tid];
if (!(evaluatePredicate(reg_customer__c_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT64f6e8a6f700, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_64f6e8a516b0(uint64_t* BUF_64f6e8a6f700, uint64_t* BUF_IDX_64f6e8a6f700, HASHTABLE_INSERT HT_64f6e8a6f700, DBI32Type* customer__c_custkey, DBStringType* customer__c_nation, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_nation = customer__c_nation[tid];
if (!(evaluatePredicate(reg_customer__c_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_64f6e8a6f700 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_64f6e8a6f700 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_64f6e8a6f700 = atomicAdd((int*)BUF_IDX_64f6e8a6f700, 1);
HT_64f6e8a6f700.insert(cuco::pair{KEY_64f6e8a6f700, buf_idx_64f6e8a6f700});
BUF_64f6e8a6f700[buf_idx_64f6e8a6f700 * 1 + 0] = tid;
}
__global__ void count_64f6e8a81170(uint64_t* COUNT64f6e8a69e50, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT64f6e8a69e50, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_64f6e8a81170(uint64_t* BUF_64f6e8a69e50, uint64_t* BUF_IDX_64f6e8a69e50, HASHTABLE_INSERT HT_64f6e8a69e50, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_64f6e8a69e50 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_64f6e8a69e50 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_64f6e8a69e50 = atomicAdd((int*)BUF_IDX_64f6e8a69e50, 1);
HT_64f6e8a69e50.insert(cuco::pair{KEY_64f6e8a69e50, buf_idx_64f6e8a69e50});
BUF_64f6e8a69e50[buf_idx_64f6e8a69e50 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_64f6e8a51c50(uint64_t* BUF_64f6e8a69e50, uint64_t* BUF_64f6e8a6cf30, uint64_t* BUF_64f6e8a6f700, HASHTABLE_INSERT HT_64f6e8a213b0, HASHTABLE_PROBE HT_64f6e8a69e50, HASHTABLE_PROBE HT_64f6e8a6cf30, HASHTABLE_PROBE HT_64f6e8a6f700, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_64f6e8a6cf30 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_64f6e8a6cf30 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_64f6e8a6cf30.for_each(KEY_64f6e8a6cf30, [&] __device__ (auto const SLOT_64f6e8a6cf30) {

auto const [slot_first64f6e8a6cf30, slot_second64f6e8a6cf30] = SLOT_64f6e8a6cf30;
if (!(true)) return;
uint64_t KEY_64f6e8a6f700 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_64f6e8a6f700 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_64f6e8a6f700.for_each(KEY_64f6e8a6f700, [&] __device__ (auto const SLOT_64f6e8a6f700) {

auto const [slot_first64f6e8a6f700, slot_second64f6e8a6f700] = SLOT_64f6e8a6f700;
if (!(true)) return;
uint64_t KEY_64f6e8a69e50 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_64f6e8a69e50 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_64f6e8a69e50.for_each(KEY_64f6e8a69e50, [&] __device__ (auto const SLOT_64f6e8a69e50) {

auto const [slot_first64f6e8a69e50, slot_second64f6e8a69e50] = SLOT_64f6e8a69e50;
if (!(true)) return;
uint64_t KEY_64f6e8a213b0 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_64f6e8a6f700[slot_second64f6e8a6f700 * 1 + 0]];

KEY_64f6e8a213b0 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_64f6e8a6cf30[slot_second64f6e8a6cf30 * 1 + 0]];
KEY_64f6e8a213b0 <<= 16;
KEY_64f6e8a213b0 |= reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[BUF_64f6e8a69e50[slot_second64f6e8a69e50 * 1 + 0]];
KEY_64f6e8a213b0 <<= 32;
KEY_64f6e8a213b0 |= reg_date__d_year;
//Create aggregation hash table
HT_64f6e8a213b0.insert(cuco::pair{KEY_64f6e8a213b0, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_64f6e8a51c50(uint64_t* BUF_64f6e8a69e50, uint64_t* BUF_64f6e8a6cf30, uint64_t* BUF_64f6e8a6f700, HASHTABLE_FIND HT_64f6e8a213b0, HASHTABLE_PROBE HT_64f6e8a69e50, HASHTABLE_PROBE HT_64f6e8a6cf30, HASHTABLE_PROBE HT_64f6e8a6f700, DBI16Type* KEY_64f6e8a213b0customer__c_city_encoded, DBI32Type* KEY_64f6e8a213b0date__d_year, DBI16Type* KEY_64f6e8a213b0supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_64f6e8a6cf30 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_64f6e8a6cf30 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_64f6e8a6cf30.for_each(KEY_64f6e8a6cf30, [&] __device__ (auto const SLOT_64f6e8a6cf30) {
auto const [slot_first64f6e8a6cf30, slot_second64f6e8a6cf30] = SLOT_64f6e8a6cf30;
if (!(true)) return;
uint64_t KEY_64f6e8a6f700 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_64f6e8a6f700 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_64f6e8a6f700.for_each(KEY_64f6e8a6f700, [&] __device__ (auto const SLOT_64f6e8a6f700) {
auto const [slot_first64f6e8a6f700, slot_second64f6e8a6f700] = SLOT_64f6e8a6f700;
if (!(true)) return;
uint64_t KEY_64f6e8a69e50 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_64f6e8a69e50 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_64f6e8a69e50.for_each(KEY_64f6e8a69e50, [&] __device__ (auto const SLOT_64f6e8a69e50) {
auto const [slot_first64f6e8a69e50, slot_second64f6e8a69e50] = SLOT_64f6e8a69e50;
if (!(true)) return;
uint64_t KEY_64f6e8a213b0 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_64f6e8a6f700[slot_second64f6e8a6f700 * 1 + 0]];

KEY_64f6e8a213b0 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_64f6e8a6cf30[slot_second64f6e8a6cf30 * 1 + 0]];
KEY_64f6e8a213b0 <<= 16;
KEY_64f6e8a213b0 |= reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[BUF_64f6e8a69e50[slot_second64f6e8a69e50 * 1 + 0]];
KEY_64f6e8a213b0 <<= 32;
KEY_64f6e8a213b0 |= reg_date__d_year;
//Aggregate in hashtable
auto buf_idx_64f6e8a213b0 = HT_64f6e8a213b0.find(KEY_64f6e8a213b0)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_64f6e8a213b0], reg_lineorder__lo_revenue);
KEY_64f6e8a213b0customer__c_city_encoded[buf_idx_64f6e8a213b0] = reg_customer__c_city_encoded;
KEY_64f6e8a213b0supplier__s_city_encoded[buf_idx_64f6e8a213b0] = reg_supplier__s_city_encoded;
KEY_64f6e8a213b0date__d_year[buf_idx_64f6e8a213b0] = reg_date__d_year;
});
});
});
}
__global__ void count_64f6e8a8b0c0(uint64_t* COUNT64f6e89fda80, size_t COUNT64f6e8a213b0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT64f6e8a213b0) return;
//Materialize count
atomicAdd((int*)COUNT64f6e89fda80, 1);
}
__global__ void main_64f6e8a8b0c0(size_t COUNT64f6e8a213b0, DBDecimalType* MAT64f6e89fda80aggr0__tmp_attr0, DBI16Type* MAT64f6e89fda80customer__c_city_encoded, DBI32Type* MAT64f6e89fda80date__d_year, DBI16Type* MAT64f6e89fda80supplier__s_city_encoded, uint64_t* MAT_IDX64f6e89fda80, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT64f6e8a213b0) return;
//Materialize buffers
auto mat_idx64f6e89fda80 = atomicAdd((int*)MAT_IDX64f6e89fda80, 1);
auto reg_customer__c_city_encoded = customer__c_city_encoded[tid];
MAT64f6e89fda80customer__c_city_encoded[mat_idx64f6e89fda80] = reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[tid];
MAT64f6e89fda80supplier__s_city_encoded[mat_idx64f6e89fda80] = reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[tid];
MAT64f6e89fda80date__d_year[mat_idx64f6e89fda80] = reg_date__d_year;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT64f6e89fda80aggr0__tmp_attr0[mat_idx64f6e89fda80] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map) {
//Materialize count
uint64_t* d_COUNT64f6e8a6cf30;
hipMalloc(&d_COUNT64f6e8a6cf30, sizeof(uint64_t));
hipMemset(d_COUNT64f6e8a6cf30, 0, sizeof(uint64_t));
count_64f6e8a78790<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT64f6e8a6cf30, d_supplier__s_nation, supplier_size);
uint64_t COUNT64f6e8a6cf30;
hipMemcpy(&COUNT64f6e8a6cf30, d_COUNT64f6e8a6cf30, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_64f6e8a6cf30;
hipMalloc(&d_BUF_IDX_64f6e8a6cf30, sizeof(uint64_t));
hipMemset(d_BUF_IDX_64f6e8a6cf30, 0, sizeof(uint64_t));
uint64_t* d_BUF_64f6e8a6cf30;
hipMalloc(&d_BUF_64f6e8a6cf30, sizeof(uint64_t) * COUNT64f6e8a6cf30 * 1);
auto d_HT_64f6e8a6cf30 = cuco::experimental::static_multimap{ (int)COUNT64f6e8a6cf30*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_64f6e8a78790<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_64f6e8a6cf30, d_BUF_IDX_64f6e8a6cf30, d_HT_64f6e8a6cf30.ref(cuco::insert), d_supplier__s_nation, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT64f6e8a6f700;
hipMalloc(&d_COUNT64f6e8a6f700, sizeof(uint64_t));
hipMemset(d_COUNT64f6e8a6f700, 0, sizeof(uint64_t));
count_64f6e8a516b0<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT64f6e8a6f700, d_customer__c_nation, customer_size);
uint64_t COUNT64f6e8a6f700;
hipMemcpy(&COUNT64f6e8a6f700, d_COUNT64f6e8a6f700, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_64f6e8a6f700;
hipMalloc(&d_BUF_IDX_64f6e8a6f700, sizeof(uint64_t));
hipMemset(d_BUF_IDX_64f6e8a6f700, 0, sizeof(uint64_t));
uint64_t* d_BUF_64f6e8a6f700;
hipMalloc(&d_BUF_64f6e8a6f700, sizeof(uint64_t) * COUNT64f6e8a6f700 * 1);
auto d_HT_64f6e8a6f700 = cuco::experimental::static_multimap{ (int)COUNT64f6e8a6f700*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_64f6e8a516b0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_64f6e8a6f700, d_BUF_IDX_64f6e8a6f700, d_HT_64f6e8a6f700.ref(cuco::insert), d_customer__c_custkey, d_customer__c_nation, customer_size);
//Materialize count
uint64_t* d_COUNT64f6e8a69e50;
hipMalloc(&d_COUNT64f6e8a69e50, sizeof(uint64_t));
hipMemset(d_COUNT64f6e8a69e50, 0, sizeof(uint64_t));
count_64f6e8a81170<<<std::ceil((float)date_size/32.), 32>>>(d_COUNT64f6e8a69e50, d_date__d_year, date_size);
uint64_t COUNT64f6e8a69e50;
hipMemcpy(&COUNT64f6e8a69e50, d_COUNT64f6e8a69e50, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_64f6e8a69e50;
hipMalloc(&d_BUF_IDX_64f6e8a69e50, sizeof(uint64_t));
hipMemset(d_BUF_IDX_64f6e8a69e50, 0, sizeof(uint64_t));
uint64_t* d_BUF_64f6e8a69e50;
hipMalloc(&d_BUF_64f6e8a69e50, sizeof(uint64_t) * COUNT64f6e8a69e50 * 1);
auto d_HT_64f6e8a69e50 = cuco::experimental::static_multimap{ (int)COUNT64f6e8a69e50*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_64f6e8a81170<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_64f6e8a69e50, d_BUF_IDX_64f6e8a69e50, d_HT_64f6e8a69e50.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Create aggregation hash table
auto d_HT_64f6e8a213b0 = cuco::static_map{ (int)5679*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_64f6e8a51c50<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_64f6e8a69e50, d_BUF_64f6e8a6cf30, d_BUF_64f6e8a6f700, d_HT_64f6e8a213b0.ref(cuco::insert), d_HT_64f6e8a69e50.ref(cuco::for_each), d_HT_64f6e8a6cf30.ref(cuco::for_each), d_HT_64f6e8a6f700.ref(cuco::for_each), d_customer__c_city_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
size_t COUNT64f6e8a213b0 = d_HT_64f6e8a213b0.size();
thrust::device_vector<int64_t> keys_64f6e8a213b0(COUNT64f6e8a213b0), vals_64f6e8a213b0(COUNT64f6e8a213b0);
d_HT_64f6e8a213b0.retrieve_all(keys_64f6e8a213b0.begin(), vals_64f6e8a213b0.begin());
d_HT_64f6e8a213b0.clear();
int64_t* raw_keys64f6e8a213b0 = thrust::raw_pointer_cast(keys_64f6e8a213b0.data());
insertKeys<<<std::ceil((float)COUNT64f6e8a213b0/32.), 32>>>(raw_keys64f6e8a213b0, d_HT_64f6e8a213b0.ref(cuco::insert), COUNT64f6e8a213b0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT64f6e8a213b0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT64f6e8a213b0);
DBI16Type* d_KEY_64f6e8a213b0customer__c_city_encoded;
hipMalloc(&d_KEY_64f6e8a213b0customer__c_city_encoded, sizeof(DBI16Type) * COUNT64f6e8a213b0);
hipMemset(d_KEY_64f6e8a213b0customer__c_city_encoded, 0, sizeof(DBI16Type) * COUNT64f6e8a213b0);
DBI16Type* d_KEY_64f6e8a213b0supplier__s_city_encoded;
hipMalloc(&d_KEY_64f6e8a213b0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT64f6e8a213b0);
hipMemset(d_KEY_64f6e8a213b0supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT64f6e8a213b0);
DBI32Type* d_KEY_64f6e8a213b0date__d_year;
hipMalloc(&d_KEY_64f6e8a213b0date__d_year, sizeof(DBI32Type) * COUNT64f6e8a213b0);
hipMemset(d_KEY_64f6e8a213b0date__d_year, 0, sizeof(DBI32Type) * COUNT64f6e8a213b0);
main_64f6e8a51c50<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_64f6e8a69e50, d_BUF_64f6e8a6cf30, d_BUF_64f6e8a6f700, d_HT_64f6e8a213b0.ref(cuco::find), d_HT_64f6e8a69e50.ref(cuco::for_each), d_HT_64f6e8a6cf30.ref(cuco::for_each), d_HT_64f6e8a6f700.ref(cuco::for_each), d_KEY_64f6e8a213b0customer__c_city_encoded, d_KEY_64f6e8a213b0date__d_year, d_KEY_64f6e8a213b0supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_city_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
//Materialize count
uint64_t* d_COUNT64f6e89fda80;
hipMalloc(&d_COUNT64f6e89fda80, sizeof(uint64_t));
hipMemset(d_COUNT64f6e89fda80, 0, sizeof(uint64_t));
count_64f6e8a8b0c0<<<std::ceil((float)COUNT64f6e8a213b0/32.), 32>>>(d_COUNT64f6e89fda80, COUNT64f6e8a213b0);
uint64_t COUNT64f6e89fda80;
hipMemcpy(&COUNT64f6e89fda80, d_COUNT64f6e89fda80, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX64f6e89fda80;
hipMalloc(&d_MAT_IDX64f6e89fda80, sizeof(uint64_t));
hipMemset(d_MAT_IDX64f6e89fda80, 0, sizeof(uint64_t));
auto MAT64f6e89fda80customer__c_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT64f6e89fda80);
DBI16Type* d_MAT64f6e89fda80customer__c_city_encoded;
hipMalloc(&d_MAT64f6e89fda80customer__c_city_encoded, sizeof(DBI16Type) * COUNT64f6e89fda80);
auto MAT64f6e89fda80supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT64f6e89fda80);
DBI16Type* d_MAT64f6e89fda80supplier__s_city_encoded;
hipMalloc(&d_MAT64f6e89fda80supplier__s_city_encoded, sizeof(DBI16Type) * COUNT64f6e89fda80);
auto MAT64f6e89fda80date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT64f6e89fda80);
DBI32Type* d_MAT64f6e89fda80date__d_year;
hipMalloc(&d_MAT64f6e89fda80date__d_year, sizeof(DBI32Type) * COUNT64f6e89fda80);
auto MAT64f6e89fda80aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT64f6e89fda80);
DBDecimalType* d_MAT64f6e89fda80aggr0__tmp_attr0;
hipMalloc(&d_MAT64f6e89fda80aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT64f6e89fda80);
main_64f6e8a8b0c0<<<std::ceil((float)COUNT64f6e8a213b0/32.), 32>>>(COUNT64f6e8a213b0, d_MAT64f6e89fda80aggr0__tmp_attr0, d_MAT64f6e89fda80customer__c_city_encoded, d_MAT64f6e89fda80date__d_year, d_MAT64f6e89fda80supplier__s_city_encoded, d_MAT_IDX64f6e89fda80, d_aggr0__tmp_attr0, d_KEY_64f6e8a213b0customer__c_city_encoded, d_KEY_64f6e8a213b0date__d_year, d_KEY_64f6e8a213b0supplier__s_city_encoded);
hipMemcpy(MAT64f6e89fda80customer__c_city_encoded, d_MAT64f6e89fda80customer__c_city_encoded, sizeof(DBI16Type) * COUNT64f6e89fda80, hipMemcpyDeviceToHost);
hipMemcpy(MAT64f6e89fda80supplier__s_city_encoded, d_MAT64f6e89fda80supplier__s_city_encoded, sizeof(DBI16Type) * COUNT64f6e89fda80, hipMemcpyDeviceToHost);
hipMemcpy(MAT64f6e89fda80date__d_year, d_MAT64f6e89fda80date__d_year, sizeof(DBI32Type) * COUNT64f6e89fda80, hipMemcpyDeviceToHost);
hipMemcpy(MAT64f6e89fda80aggr0__tmp_attr0, d_MAT64f6e89fda80aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT64f6e89fda80, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT64f6e89fda80; i++) { std::cout << customer__c_city_map[MAT64f6e89fda80customer__c_city_encoded[i]] << "\t";
std::cout << supplier__s_city_map[MAT64f6e89fda80supplier__s_city_encoded[i]] << "\t";
std::cout << MAT64f6e89fda80date__d_year[i] << "\t";
std::cout << MAT64f6e89fda80aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_64f6e8a6cf30);
hipFree(d_BUF_IDX_64f6e8a6cf30);
hipFree(d_COUNT64f6e8a6cf30);
hipFree(d_BUF_64f6e8a6f700);
hipFree(d_BUF_IDX_64f6e8a6f700);
hipFree(d_COUNT64f6e8a6f700);
hipFree(d_BUF_64f6e8a69e50);
hipFree(d_BUF_IDX_64f6e8a69e50);
hipFree(d_COUNT64f6e8a69e50);
hipFree(d_KEY_64f6e8a213b0customer__c_city_encoded);
hipFree(d_KEY_64f6e8a213b0date__d_year);
hipFree(d_KEY_64f6e8a213b0supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT64f6e89fda80);
hipFree(d_MAT64f6e89fda80aggr0__tmp_attr0);
hipFree(d_MAT64f6e89fda80customer__c_city_encoded);
hipFree(d_MAT64f6e89fda80date__d_year);
hipFree(d_MAT64f6e89fda80supplier__s_city_encoded);
hipFree(d_MAT_IDX64f6e89fda80);
free(MAT64f6e89fda80aggr0__tmp_attr0);
free(MAT64f6e89fda80customer__c_city_encoded);
free(MAT64f6e89fda80date__d_year);
free(MAT64f6e89fda80supplier__s_city_encoded);
}