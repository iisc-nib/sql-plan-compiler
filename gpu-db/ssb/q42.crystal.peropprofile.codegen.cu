#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_11(uint64_t* COUNT29, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_supplier__s_region[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_region[ITEM] = supplier__s_region[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_supplier__s_region[ITEM], "AMERICA", Predicate::eq);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT29, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_11(uint64_t* BUF_29, uint64_t* BUF_IDX_29, HASHTABLE_INSERT HT_29, int64_t* cycles_per_warp_main_11_join_build_29, int64_t* cycles_per_warp_main_11_selection_10, int64_t* cycles_per_warp_main_11_selection_12, int64_t* cycles_per_warp_main_11_selection_13, int64_t* cycles_per_warp_main_11_selection_14, int64_t* cycles_per_warp_main_11_selection_15, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_supplier__s_region[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_region[ITEM] = supplier__s_region[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_supplier__s_region[ITEM], "AMERICA", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_selection_10[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_selection_12[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_selection_13[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_selection_14[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_selection_15[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_29[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_29[ITEM] = 0;
KEY_29[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_29 = atomicAdd((int*)BUF_IDX_29, 1);
HT_29.insert(cuco::pair{KEY_29[ITEM], buf_idx_29});
BUF_29[(buf_idx_29) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_join_build_29[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_5(uint64_t* COUNT30, DBStringType* customer__c_region, size_t customer_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_customer__c_region[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_region[ITEM] = customer__c_region[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_customer__c_region[ITEM], "AMERICA", Predicate::eq);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT30, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_30, uint64_t* BUF_IDX_30, HASHTABLE_INSERT HT_30, DBI32Type* customer__c_custkey, DBStringType* customer__c_region, size_t customer_size, int64_t* cycles_per_warp_main_5_join_build_30, int64_t* cycles_per_warp_main_5_selection_4, int64_t* cycles_per_warp_main_5_selection_6, int64_t* cycles_per_warp_main_5_selection_7, int64_t* cycles_per_warp_main_5_selection_8, int64_t* cycles_per_warp_main_5_selection_9) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_customer__c_region[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_region[ITEM] = customer__c_region[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_customer__c_region[ITEM], "AMERICA", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_selection_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_selection_6[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_selection_7[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_selection_8[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_selection_9[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_30[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_30[ITEM] = 0;
KEY_30[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_30 = atomicAdd((int*)BUF_IDX_30, 1);
HT_30.insert(cuco::pair{KEY_30[ITEM], buf_idx_30});
BUF_30[(buf_idx_30) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_join_build_30[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_1(uint64_t* COUNT31, DBI32Type* date__d_year, size_t date_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_year[ITEM] = date__d_year[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_date__d_year[ITEM], 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year[ITEM], 1998, Predicate::eq));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT31, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_31, uint64_t* BUF_IDX_31, HASHTABLE_INSERT HT_31, int64_t* cycles_per_warp_main_1_join_build_31, int64_t* cycles_per_warp_main_1_selection_0, int64_t* cycles_per_warp_main_1_selection_2, int64_t* cycles_per_warp_main_1_selection_3, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_year[ITEM] = date__d_year[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_date__d_year[ITEM], 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year[ITEM], 1998, Predicate::eq));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_3[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_31[ITEMS_PER_THREAD];
DBI32Type reg_date__d_datekey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_datekey[ITEM] = date__d_datekey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_31[ITEM] = 0;
KEY_31[ITEM] |= reg_date__d_datekey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_31 = atomicAdd((int*)BUF_IDX_31, 1);
HT_31.insert(cuco::pair{KEY_31[ITEM], buf_idx_31});
BUF_31[(buf_idx_31) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_build_31[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_17(uint64_t* COUNT32, DBStringType* part__p_mfgr, size_t part_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_part__p_mfgr[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_mfgr[ITEM] = part__p_mfgr[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_part__p_mfgr[ITEM], "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr[ITEM], "MFGR#2", Predicate::eq));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT32, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_17(uint64_t* BUF_32, uint64_t* BUF_IDX_32, HASHTABLE_INSERT HT_32, int64_t* cycles_per_warp_main_17_join_build_32, int64_t* cycles_per_warp_main_17_selection_16, int64_t* cycles_per_warp_main_17_selection_18, int64_t* cycles_per_warp_main_17_selection_19, DBStringType* part__p_mfgr, DBI32Type* part__p_partkey, size_t part_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_part__p_mfgr[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_mfgr[ITEM] = part__p_mfgr[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_part__p_mfgr[ITEM], "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr[ITEM], "MFGR#2", Predicate::eq));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_selection_16[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_selection_18[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_selection_19[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_32[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_32[ITEM] = 0;
KEY_32[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_32 = atomicAdd((int*)BUF_IDX_32, 1);
HT_32.insert(cuco::pair{KEY_32[ITEM], buf_idx_32});
BUF_32[(buf_idx_32) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_join_build_32[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_21(uint64_t* BUF_29, uint64_t* BUF_30, uint64_t* BUF_31, uint64_t* BUF_32, HASHTABLE_PROBE HT_29, HASHTABLE_PROBE HT_30, HASHTABLE_PROBE HT_31, HASHTABLE_PROBE HT_32, HASHTABLE_INSERT HT_34, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* part__p_category_encoded, DBI16Type* supplier__s_nation_encoded) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
uint64_t KEY_29[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_suppkey[ITEM] = lineorder__lo_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_29[ITEM] = 0;
KEY_29[ITEM] |= reg_lineorder__lo_suppkey[ITEM];
}
int64_t slot_second29[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_29 = HT_29.find(KEY_29[ITEM]);
if (SLOT_29 == HT_29.end()) {selection_flags[ITEM] = 0; continue;}
slot_second29[ITEM] = SLOT_29->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_30[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_custkey[ITEM] = lineorder__lo_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_30[ITEM] = 0;
KEY_30[ITEM] |= reg_lineorder__lo_custkey[ITEM];
}
int64_t slot_second30[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_30 = HT_30.find(KEY_30[ITEM]);
if (SLOT_30 == HT_30.end()) {selection_flags[ITEM] = 0; continue;}
slot_second30[ITEM] = SLOT_30->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_31[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_orderdate[ITEM] = lineorder__lo_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_31[ITEM] = 0;
KEY_31[ITEM] |= reg_lineorder__lo_orderdate[ITEM];
}
int64_t slot_second31[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_31 = HT_31.find(KEY_31[ITEM]);
if (SLOT_31 == HT_31.end()) {selection_flags[ITEM] = 0; continue;}
slot_second31[ITEM] = SLOT_31->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_32[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_partkey[ITEM] = lineorder__lo_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_32[ITEM] = 0;
KEY_32[ITEM] |= reg_lineorder__lo_partkey[ITEM];
}
int64_t slot_second32[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_32 = HT_32.find(KEY_32[ITEM]);
if (SLOT_32 == HT_32.end()) {selection_flags[ITEM] = 0; continue;}
slot_second32[ITEM] = SLOT_32->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_34[ITEMS_PER_THREAD];
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_date__d_year[ITEM] = date__d_year[BUF_31[slot_second31[ITEM] * 1 + 0]];
}
DBI16Type reg_supplier__s_nation_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_supplier__s_nation_encoded[ITEM] = supplier__s_nation_encoded[BUF_29[slot_second29[ITEM] * 1 + 0]];
}
DBI16Type reg_part__p_category_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_category_encoded[ITEM] = part__p_category_encoded[BUF_32[slot_second32[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_34[ITEM] = 0;
KEY_34[ITEM] |= reg_date__d_year[ITEM];
KEY_34[ITEM] <<= 16;
KEY_34[ITEM] |= reg_supplier__s_nation_encoded[ITEM];
KEY_34[ITEM] <<= 16;
KEY_34[ITEM] |= reg_part__p_category_encoded[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_34.insert(cuco::pair{KEY_34[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_21(uint64_t* BUF_29, uint64_t* BUF_30, uint64_t* BUF_31, uint64_t* BUF_32, HASHTABLE_PROBE HT_29, HASHTABLE_PROBE HT_30, HASHTABLE_PROBE HT_31, HASHTABLE_PROBE HT_32, HASHTABLE_FIND HT_34, DBI32Type* KEY_34date__d_year, DBI16Type* KEY_34part__p_category_encoded, DBI16Type* KEY_34supplier__s_nation_encoded, DBDecimalType* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_21_aggregation_34, int64_t* cycles_per_warp_main_21_join_probe_29, int64_t* cycles_per_warp_main_21_join_probe_30, int64_t* cycles_per_warp_main_21_join_probe_31, int64_t* cycles_per_warp_main_21_join_probe_32, int64_t* cycles_per_warp_main_21_map_33, int64_t* cycles_per_warp_main_21_selection_20, int64_t* cycles_per_warp_main_21_selection_22, int64_t* cycles_per_warp_main_21_selection_23, int64_t* cycles_per_warp_main_21_selection_24, int64_t* cycles_per_warp_main_21_selection_25, int64_t* cycles_per_warp_main_21_selection_26, int64_t* cycles_per_warp_main_21_selection_27, int64_t* cycles_per_warp_main_21_selection_28, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, DBDecimalType* lineorder__lo_supplycost, size_t lineorder_size, DBI16Type* part__p_category_encoded, DBI16Type* supplier__s_nation_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_21_selection_20[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_21_selection_22[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_21_selection_23[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_21_selection_24[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_21_selection_25[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_21_selection_26[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_21_selection_27[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_21_selection_28[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_29[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_suppkey[ITEM] = lineorder__lo_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_29[ITEM] = 0;
KEY_29[ITEM] |= reg_lineorder__lo_suppkey[ITEM];
}
int64_t slot_second29[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_29 = HT_29.find(KEY_29[ITEM]);
if (SLOT_29 == HT_29.end()) {selection_flags[ITEM] = 0; continue;}
slot_second29[ITEM] = SLOT_29->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_21_join_probe_29[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_30[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_custkey[ITEM] = lineorder__lo_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_30[ITEM] = 0;
KEY_30[ITEM] |= reg_lineorder__lo_custkey[ITEM];
}
int64_t slot_second30[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_30 = HT_30.find(KEY_30[ITEM]);
if (SLOT_30 == HT_30.end()) {selection_flags[ITEM] = 0; continue;}
slot_second30[ITEM] = SLOT_30->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_21_join_probe_30[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_31[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_orderdate[ITEM] = lineorder__lo_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_31[ITEM] = 0;
KEY_31[ITEM] |= reg_lineorder__lo_orderdate[ITEM];
}
int64_t slot_second31[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_31 = HT_31.find(KEY_31[ITEM]);
if (SLOT_31 == HT_31.end()) {selection_flags[ITEM] = 0; continue;}
slot_second31[ITEM] = SLOT_31->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_21_join_probe_31[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_32[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_partkey[ITEM] = lineorder__lo_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_32[ITEM] = 0;
KEY_32[ITEM] |= reg_lineorder__lo_partkey[ITEM];
}
int64_t slot_second32[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_32 = HT_32.find(KEY_32[ITEM]);
if (SLOT_32 == HT_32.end()) {selection_flags[ITEM] = 0; continue;}
slot_second32[ITEM] = SLOT_32->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_21_join_probe_32[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_21_map_33[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_34[ITEMS_PER_THREAD];
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_date__d_year[ITEM] = date__d_year[BUF_31[slot_second31[ITEM] * 1 + 0]];
}
DBI16Type reg_supplier__s_nation_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_supplier__s_nation_encoded[ITEM] = supplier__s_nation_encoded[BUF_29[slot_second29[ITEM] * 1 + 0]];
}
DBI16Type reg_part__p_category_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_category_encoded[ITEM] = part__p_category_encoded[BUF_32[slot_second32[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_34[ITEM] = 0;
KEY_34[ITEM] |= reg_date__d_year[ITEM];
KEY_34[ITEM] <<= 16;
KEY_34[ITEM] |= reg_supplier__s_nation_encoded[ITEM];
KEY_34[ITEM] <<= 16;
KEY_34[ITEM] |= reg_part__p_category_encoded[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineorder__lo_supplycost[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_supplycost[ITEM] = lineorder__lo_supplycost[ITEM*TB + tid];
}
DBDecimalType reg_lineorder__lo_revenue[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_revenue[ITEM] = lineorder__lo_revenue[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineorder__lo_revenue[ITEM]) - (reg_lineorder__lo_supplycost[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_34 = HT_34.find(KEY_34[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr0[buf_idx_34], reg_map0__tmp_attr1[ITEM]);
KEY_34date__d_year[buf_idx_34] = reg_date__d_year[ITEM];
KEY_34supplier__s_nation_encoded[buf_idx_34] = reg_supplier__s_nation_encoded[ITEM];
KEY_34part__p_category_encoded[buf_idx_34] = reg_part__p_category_encoded[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_21_aggregation_34[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_36(size_t COUNT34, uint64_t* COUNT35) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT34); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT35, 1);
}
}
__global__ void main_36(size_t COUNT34, DBDecimalType* MAT35aggr0__tmp_attr0, DBI32Type* MAT35date__d_year, DBI16Type* MAT35part__p_category_encoded, DBI16Type* MAT35supplier__s_nation_encoded, uint64_t* MAT_IDX35, DBDecimalType* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_36_materialize_35, DBI32Type* date__d_year, DBI16Type* part__p_category_encoded, DBI16Type* supplier__s_nation_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT34); ++ITEM) {
reg_date__d_year[ITEM] = date__d_year[ITEM*TB + tid];
}
DBI16Type reg_supplier__s_nation_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT34); ++ITEM) {
reg_supplier__s_nation_encoded[ITEM] = supplier__s_nation_encoded[ITEM*TB + tid];
}
DBI16Type reg_part__p_category_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT34); ++ITEM) {
reg_part__p_category_encoded[ITEM] = part__p_category_encoded[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT34); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT34); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx35 = atomicAdd((int*)MAT_IDX35, 1);
MAT35date__d_year[mat_idx35] = reg_date__d_year[ITEM];
MAT35supplier__s_nation_encoded[mat_idx35] = reg_supplier__s_nation_encoded[ITEM];
MAT35part__p_category_encoded[mat_idx35] = reg_part__p_category_encoded[ITEM];
MAT35aggr0__tmp_attr0[mat_idx35] = reg_aggr0__tmp_attr0[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_36_materialize_35[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_11_selection_10;
auto main_11_selection_10_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_selection_10, sizeof(int64_t) * main_11_selection_10_cpw_size);
hipMemset(d_cycles_per_warp_main_11_selection_10, -1, sizeof(int64_t) * main_11_selection_10_cpw_size);
int64_t* d_cycles_per_warp_main_11_selection_12;
auto main_11_selection_12_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_selection_12, sizeof(int64_t) * main_11_selection_12_cpw_size);
hipMemset(d_cycles_per_warp_main_11_selection_12, -1, sizeof(int64_t) * main_11_selection_12_cpw_size);
int64_t* d_cycles_per_warp_main_11_selection_13;
auto main_11_selection_13_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_selection_13, sizeof(int64_t) * main_11_selection_13_cpw_size);
hipMemset(d_cycles_per_warp_main_11_selection_13, -1, sizeof(int64_t) * main_11_selection_13_cpw_size);
int64_t* d_cycles_per_warp_main_11_selection_14;
auto main_11_selection_14_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_selection_14, sizeof(int64_t) * main_11_selection_14_cpw_size);
hipMemset(d_cycles_per_warp_main_11_selection_14, -1, sizeof(int64_t) * main_11_selection_14_cpw_size);
int64_t* d_cycles_per_warp_main_11_selection_15;
auto main_11_selection_15_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_selection_15, sizeof(int64_t) * main_11_selection_15_cpw_size);
hipMemset(d_cycles_per_warp_main_11_selection_15, -1, sizeof(int64_t) * main_11_selection_15_cpw_size);
//Materialize count
uint64_t* d_COUNT29;
hipMalloc(&d_COUNT29, sizeof(uint64_t));
hipMemset(d_COUNT29, 0, sizeof(uint64_t));
count_11<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT29, d_supplier__s_region, supplier_size);
uint64_t COUNT29;
hipMemcpy(&COUNT29, d_COUNT29, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_11_join_build_29;
auto main_11_join_build_29_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_join_build_29, sizeof(int64_t) * main_11_join_build_29_cpw_size);
hipMemset(d_cycles_per_warp_main_11_join_build_29, -1, sizeof(int64_t) * main_11_join_build_29_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_29;
hipMalloc(&d_BUF_IDX_29, sizeof(uint64_t));
hipMemset(d_BUF_IDX_29, 0, sizeof(uint64_t));
uint64_t* d_BUF_29;
hipMalloc(&d_BUF_29, sizeof(uint64_t) * COUNT29 * 1);
auto d_HT_29 = cuco::static_map{ (int)COUNT29*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_11<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_29, d_BUF_IDX_29, d_HT_29.ref(cuco::insert), d_cycles_per_warp_main_11_join_build_29, d_cycles_per_warp_main_11_selection_10, d_cycles_per_warp_main_11_selection_12, d_cycles_per_warp_main_11_selection_13, d_cycles_per_warp_main_11_selection_14, d_cycles_per_warp_main_11_selection_15, d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
int64_t* cycles_per_warp_main_11_selection_10 = (int64_t*)malloc(sizeof(int64_t) * main_11_selection_10_cpw_size);
hipMemcpy(cycles_per_warp_main_11_selection_10, d_cycles_per_warp_main_11_selection_10, sizeof(int64_t) * main_11_selection_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_selection_10 ";
for (auto i=0ull; i < main_11_selection_10_cpw_size; i++) std::cout << cycles_per_warp_main_11_selection_10[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_11_selection_12 = (int64_t*)malloc(sizeof(int64_t) * main_11_selection_12_cpw_size);
hipMemcpy(cycles_per_warp_main_11_selection_12, d_cycles_per_warp_main_11_selection_12, sizeof(int64_t) * main_11_selection_12_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_selection_12 ";
for (auto i=0ull; i < main_11_selection_12_cpw_size; i++) std::cout << cycles_per_warp_main_11_selection_12[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_11_selection_13 = (int64_t*)malloc(sizeof(int64_t) * main_11_selection_13_cpw_size);
hipMemcpy(cycles_per_warp_main_11_selection_13, d_cycles_per_warp_main_11_selection_13, sizeof(int64_t) * main_11_selection_13_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_selection_13 ";
for (auto i=0ull; i < main_11_selection_13_cpw_size; i++) std::cout << cycles_per_warp_main_11_selection_13[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_11_selection_14 = (int64_t*)malloc(sizeof(int64_t) * main_11_selection_14_cpw_size);
hipMemcpy(cycles_per_warp_main_11_selection_14, d_cycles_per_warp_main_11_selection_14, sizeof(int64_t) * main_11_selection_14_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_selection_14 ";
for (auto i=0ull; i < main_11_selection_14_cpw_size; i++) std::cout << cycles_per_warp_main_11_selection_14[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_11_selection_15 = (int64_t*)malloc(sizeof(int64_t) * main_11_selection_15_cpw_size);
hipMemcpy(cycles_per_warp_main_11_selection_15, d_cycles_per_warp_main_11_selection_15, sizeof(int64_t) * main_11_selection_15_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_selection_15 ";
for (auto i=0ull; i < main_11_selection_15_cpw_size; i++) std::cout << cycles_per_warp_main_11_selection_15[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_11_join_build_29 = (int64_t*)malloc(sizeof(int64_t) * main_11_join_build_29_cpw_size);
hipMemcpy(cycles_per_warp_main_11_join_build_29, d_cycles_per_warp_main_11_join_build_29, sizeof(int64_t) * main_11_join_build_29_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_join_build_29 ";
for (auto i=0ull; i < main_11_join_build_29_cpw_size; i++) std::cout << cycles_per_warp_main_11_join_build_29[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_5_selection_4;
auto main_5_selection_4_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_selection_4, sizeof(int64_t) * main_5_selection_4_cpw_size);
hipMemset(d_cycles_per_warp_main_5_selection_4, -1, sizeof(int64_t) * main_5_selection_4_cpw_size);
int64_t* d_cycles_per_warp_main_5_selection_6;
auto main_5_selection_6_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_selection_6, sizeof(int64_t) * main_5_selection_6_cpw_size);
hipMemset(d_cycles_per_warp_main_5_selection_6, -1, sizeof(int64_t) * main_5_selection_6_cpw_size);
int64_t* d_cycles_per_warp_main_5_selection_7;
auto main_5_selection_7_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_selection_7, sizeof(int64_t) * main_5_selection_7_cpw_size);
hipMemset(d_cycles_per_warp_main_5_selection_7, -1, sizeof(int64_t) * main_5_selection_7_cpw_size);
int64_t* d_cycles_per_warp_main_5_selection_8;
auto main_5_selection_8_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_selection_8, sizeof(int64_t) * main_5_selection_8_cpw_size);
hipMemset(d_cycles_per_warp_main_5_selection_8, -1, sizeof(int64_t) * main_5_selection_8_cpw_size);
int64_t* d_cycles_per_warp_main_5_selection_9;
auto main_5_selection_9_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_selection_9, sizeof(int64_t) * main_5_selection_9_cpw_size);
hipMemset(d_cycles_per_warp_main_5_selection_9, -1, sizeof(int64_t) * main_5_selection_9_cpw_size);
//Materialize count
uint64_t* d_COUNT30;
hipMalloc(&d_COUNT30, sizeof(uint64_t));
hipMemset(d_COUNT30, 0, sizeof(uint64_t));
count_5<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT30, d_customer__c_region, customer_size);
uint64_t COUNT30;
hipMemcpy(&COUNT30, d_COUNT30, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_5_join_build_30;
auto main_5_join_build_30_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_join_build_30, sizeof(int64_t) * main_5_join_build_30_cpw_size);
hipMemset(d_cycles_per_warp_main_5_join_build_30, -1, sizeof(int64_t) * main_5_join_build_30_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_30;
hipMalloc(&d_BUF_IDX_30, sizeof(uint64_t));
hipMemset(d_BUF_IDX_30, 0, sizeof(uint64_t));
uint64_t* d_BUF_30;
hipMalloc(&d_BUF_30, sizeof(uint64_t) * COUNT30 * 1);
auto d_HT_30 = cuco::static_map{ (int)COUNT30*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_30, d_BUF_IDX_30, d_HT_30.ref(cuco::insert), d_customer__c_custkey, d_customer__c_region, customer_size, d_cycles_per_warp_main_5_join_build_30, d_cycles_per_warp_main_5_selection_4, d_cycles_per_warp_main_5_selection_6, d_cycles_per_warp_main_5_selection_7, d_cycles_per_warp_main_5_selection_8, d_cycles_per_warp_main_5_selection_9);
int64_t* cycles_per_warp_main_5_selection_4 = (int64_t*)malloc(sizeof(int64_t) * main_5_selection_4_cpw_size);
hipMemcpy(cycles_per_warp_main_5_selection_4, d_cycles_per_warp_main_5_selection_4, sizeof(int64_t) * main_5_selection_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_selection_4 ";
for (auto i=0ull; i < main_5_selection_4_cpw_size; i++) std::cout << cycles_per_warp_main_5_selection_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_selection_6 = (int64_t*)malloc(sizeof(int64_t) * main_5_selection_6_cpw_size);
hipMemcpy(cycles_per_warp_main_5_selection_6, d_cycles_per_warp_main_5_selection_6, sizeof(int64_t) * main_5_selection_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_selection_6 ";
for (auto i=0ull; i < main_5_selection_6_cpw_size; i++) std::cout << cycles_per_warp_main_5_selection_6[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_selection_7 = (int64_t*)malloc(sizeof(int64_t) * main_5_selection_7_cpw_size);
hipMemcpy(cycles_per_warp_main_5_selection_7, d_cycles_per_warp_main_5_selection_7, sizeof(int64_t) * main_5_selection_7_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_selection_7 ";
for (auto i=0ull; i < main_5_selection_7_cpw_size; i++) std::cout << cycles_per_warp_main_5_selection_7[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_selection_8 = (int64_t*)malloc(sizeof(int64_t) * main_5_selection_8_cpw_size);
hipMemcpy(cycles_per_warp_main_5_selection_8, d_cycles_per_warp_main_5_selection_8, sizeof(int64_t) * main_5_selection_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_selection_8 ";
for (auto i=0ull; i < main_5_selection_8_cpw_size; i++) std::cout << cycles_per_warp_main_5_selection_8[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_selection_9 = (int64_t*)malloc(sizeof(int64_t) * main_5_selection_9_cpw_size);
hipMemcpy(cycles_per_warp_main_5_selection_9, d_cycles_per_warp_main_5_selection_9, sizeof(int64_t) * main_5_selection_9_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_selection_9 ";
for (auto i=0ull; i < main_5_selection_9_cpw_size; i++) std::cout << cycles_per_warp_main_5_selection_9[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_join_build_30 = (int64_t*)malloc(sizeof(int64_t) * main_5_join_build_30_cpw_size);
hipMemcpy(cycles_per_warp_main_5_join_build_30, d_cycles_per_warp_main_5_join_build_30, sizeof(int64_t) * main_5_join_build_30_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_join_build_30 ";
for (auto i=0ull; i < main_5_join_build_30_cpw_size; i++) std::cout << cycles_per_warp_main_5_join_build_30[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_2;
auto main_1_selection_2_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_2, -1, sizeof(int64_t) * main_1_selection_2_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_3;
auto main_1_selection_3_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_3, -1, sizeof(int64_t) * main_1_selection_3_cpw_size);
//Materialize count
uint64_t* d_COUNT31;
hipMalloc(&d_COUNT31, sizeof(uint64_t));
hipMemset(d_COUNT31, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)date_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT31, d_date__d_year, date_size);
uint64_t COUNT31;
hipMemcpy(&COUNT31, d_COUNT31, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_1_join_build_31;
auto main_1_join_build_31_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_build_31, sizeof(int64_t) * main_1_join_build_31_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_build_31, -1, sizeof(int64_t) * main_1_join_build_31_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_31;
hipMalloc(&d_BUF_IDX_31, sizeof(uint64_t));
hipMemset(d_BUF_IDX_31, 0, sizeof(uint64_t));
uint64_t* d_BUF_31;
hipMalloc(&d_BUF_31, sizeof(uint64_t) * COUNT31 * 1);
auto d_HT_31 = cuco::static_map{ (int)COUNT31*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)date_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_31, d_BUF_IDX_31, d_HT_31.ref(cuco::insert), d_cycles_per_warp_main_1_join_build_31, d_cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_3, d_date__d_datekey, d_date__d_year, date_size);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_2 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_2 ";
for (auto i=0ull; i < main_1_selection_2_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_3 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_3, d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_3 ";
for (auto i=0ull; i < main_1_selection_3_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_3[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_build_31 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_build_31_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_build_31, d_cycles_per_warp_main_1_join_build_31, sizeof(int64_t) * main_1_join_build_31_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_build_31 ";
for (auto i=0ull; i < main_1_join_build_31_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_build_31[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_17_selection_16;
auto main_17_selection_16_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_selection_16, sizeof(int64_t) * main_17_selection_16_cpw_size);
hipMemset(d_cycles_per_warp_main_17_selection_16, -1, sizeof(int64_t) * main_17_selection_16_cpw_size);
int64_t* d_cycles_per_warp_main_17_selection_18;
auto main_17_selection_18_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_selection_18, sizeof(int64_t) * main_17_selection_18_cpw_size);
hipMemset(d_cycles_per_warp_main_17_selection_18, -1, sizeof(int64_t) * main_17_selection_18_cpw_size);
int64_t* d_cycles_per_warp_main_17_selection_19;
auto main_17_selection_19_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_selection_19, sizeof(int64_t) * main_17_selection_19_cpw_size);
hipMemset(d_cycles_per_warp_main_17_selection_19, -1, sizeof(int64_t) * main_17_selection_19_cpw_size);
//Materialize count
uint64_t* d_COUNT32;
hipMalloc(&d_COUNT32, sizeof(uint64_t));
hipMemset(d_COUNT32, 0, sizeof(uint64_t));
count_17<<<std::ceil((float)part_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT32, d_part__p_mfgr, part_size);
uint64_t COUNT32;
hipMemcpy(&COUNT32, d_COUNT32, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_17_join_build_32;
auto main_17_join_build_32_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_join_build_32, sizeof(int64_t) * main_17_join_build_32_cpw_size);
hipMemset(d_cycles_per_warp_main_17_join_build_32, -1, sizeof(int64_t) * main_17_join_build_32_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_32;
hipMalloc(&d_BUF_IDX_32, sizeof(uint64_t));
hipMemset(d_BUF_IDX_32, 0, sizeof(uint64_t));
uint64_t* d_BUF_32;
hipMalloc(&d_BUF_32, sizeof(uint64_t) * COUNT32 * 1);
auto d_HT_32 = cuco::static_map{ (int)COUNT32*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_17<<<std::ceil((float)part_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_32, d_BUF_IDX_32, d_HT_32.ref(cuco::insert), d_cycles_per_warp_main_17_join_build_32, d_cycles_per_warp_main_17_selection_16, d_cycles_per_warp_main_17_selection_18, d_cycles_per_warp_main_17_selection_19, d_part__p_mfgr, d_part__p_partkey, part_size);
int64_t* cycles_per_warp_main_17_selection_16 = (int64_t*)malloc(sizeof(int64_t) * main_17_selection_16_cpw_size);
hipMemcpy(cycles_per_warp_main_17_selection_16, d_cycles_per_warp_main_17_selection_16, sizeof(int64_t) * main_17_selection_16_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_selection_16 ";
for (auto i=0ull; i < main_17_selection_16_cpw_size; i++) std::cout << cycles_per_warp_main_17_selection_16[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_17_selection_18 = (int64_t*)malloc(sizeof(int64_t) * main_17_selection_18_cpw_size);
hipMemcpy(cycles_per_warp_main_17_selection_18, d_cycles_per_warp_main_17_selection_18, sizeof(int64_t) * main_17_selection_18_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_selection_18 ";
for (auto i=0ull; i < main_17_selection_18_cpw_size; i++) std::cout << cycles_per_warp_main_17_selection_18[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_17_selection_19 = (int64_t*)malloc(sizeof(int64_t) * main_17_selection_19_cpw_size);
hipMemcpy(cycles_per_warp_main_17_selection_19, d_cycles_per_warp_main_17_selection_19, sizeof(int64_t) * main_17_selection_19_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_selection_19 ";
for (auto i=0ull; i < main_17_selection_19_cpw_size; i++) std::cout << cycles_per_warp_main_17_selection_19[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_17_join_build_32 = (int64_t*)malloc(sizeof(int64_t) * main_17_join_build_32_cpw_size);
hipMemcpy(cycles_per_warp_main_17_join_build_32, d_cycles_per_warp_main_17_join_build_32, sizeof(int64_t) * main_17_join_build_32_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_join_build_32 ";
for (auto i=0ull; i < main_17_join_build_32_cpw_size; i++) std::cout << cycles_per_warp_main_17_join_build_32[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_21_selection_20;
auto main_21_selection_20_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_21_selection_20, sizeof(int64_t) * main_21_selection_20_cpw_size);
hipMemset(d_cycles_per_warp_main_21_selection_20, -1, sizeof(int64_t) * main_21_selection_20_cpw_size);
int64_t* d_cycles_per_warp_main_21_selection_22;
auto main_21_selection_22_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_21_selection_22, sizeof(int64_t) * main_21_selection_22_cpw_size);
hipMemset(d_cycles_per_warp_main_21_selection_22, -1, sizeof(int64_t) * main_21_selection_22_cpw_size);
int64_t* d_cycles_per_warp_main_21_selection_23;
auto main_21_selection_23_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_21_selection_23, sizeof(int64_t) * main_21_selection_23_cpw_size);
hipMemset(d_cycles_per_warp_main_21_selection_23, -1, sizeof(int64_t) * main_21_selection_23_cpw_size);
int64_t* d_cycles_per_warp_main_21_selection_24;
auto main_21_selection_24_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_21_selection_24, sizeof(int64_t) * main_21_selection_24_cpw_size);
hipMemset(d_cycles_per_warp_main_21_selection_24, -1, sizeof(int64_t) * main_21_selection_24_cpw_size);
int64_t* d_cycles_per_warp_main_21_selection_25;
auto main_21_selection_25_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_21_selection_25, sizeof(int64_t) * main_21_selection_25_cpw_size);
hipMemset(d_cycles_per_warp_main_21_selection_25, -1, sizeof(int64_t) * main_21_selection_25_cpw_size);
int64_t* d_cycles_per_warp_main_21_selection_26;
auto main_21_selection_26_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_21_selection_26, sizeof(int64_t) * main_21_selection_26_cpw_size);
hipMemset(d_cycles_per_warp_main_21_selection_26, -1, sizeof(int64_t) * main_21_selection_26_cpw_size);
int64_t* d_cycles_per_warp_main_21_selection_27;
auto main_21_selection_27_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_21_selection_27, sizeof(int64_t) * main_21_selection_27_cpw_size);
hipMemset(d_cycles_per_warp_main_21_selection_27, -1, sizeof(int64_t) * main_21_selection_27_cpw_size);
int64_t* d_cycles_per_warp_main_21_selection_28;
auto main_21_selection_28_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_21_selection_28, sizeof(int64_t) * main_21_selection_28_cpw_size);
hipMemset(d_cycles_per_warp_main_21_selection_28, -1, sizeof(int64_t) * main_21_selection_28_cpw_size);
int64_t* d_cycles_per_warp_main_21_join_probe_29;
auto main_21_join_probe_29_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_21_join_probe_29, sizeof(int64_t) * main_21_join_probe_29_cpw_size);
hipMemset(d_cycles_per_warp_main_21_join_probe_29, -1, sizeof(int64_t) * main_21_join_probe_29_cpw_size);
int64_t* d_cycles_per_warp_main_21_join_probe_30;
auto main_21_join_probe_30_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_21_join_probe_30, sizeof(int64_t) * main_21_join_probe_30_cpw_size);
hipMemset(d_cycles_per_warp_main_21_join_probe_30, -1, sizeof(int64_t) * main_21_join_probe_30_cpw_size);
int64_t* d_cycles_per_warp_main_21_join_probe_31;
auto main_21_join_probe_31_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_21_join_probe_31, sizeof(int64_t) * main_21_join_probe_31_cpw_size);
hipMemset(d_cycles_per_warp_main_21_join_probe_31, -1, sizeof(int64_t) * main_21_join_probe_31_cpw_size);
int64_t* d_cycles_per_warp_main_21_join_probe_32;
auto main_21_join_probe_32_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_21_join_probe_32, sizeof(int64_t) * main_21_join_probe_32_cpw_size);
hipMemset(d_cycles_per_warp_main_21_join_probe_32, -1, sizeof(int64_t) * main_21_join_probe_32_cpw_size);
int64_t* d_cycles_per_warp_main_21_map_33;
auto main_21_map_33_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_21_map_33, sizeof(int64_t) * main_21_map_33_cpw_size);
hipMemset(d_cycles_per_warp_main_21_map_33, -1, sizeof(int64_t) * main_21_map_33_cpw_size);
//Create aggregation hash table
auto d_HT_34 = cuco::static_map{ (int)24650*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_21<<<std::ceil((float)lineorder_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_29, d_BUF_30, d_BUF_31, d_BUF_32, d_HT_29.ref(cuco::find), d_HT_30.ref(cuco::find), d_HT_31.ref(cuco::find), d_HT_32.ref(cuco::find), d_HT_34.ref(cuco::insert), d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size, d_part__p_category_encoded, d_supplier__s_nation_encoded);
size_t COUNT34 = d_HT_34.size();
thrust::device_vector<int64_t> keys_34(COUNT34), vals_34(COUNT34);
d_HT_34.retrieve_all(keys_34.begin(), vals_34.begin());
d_HT_34.clear();
int64_t* raw_keys34 = thrust::raw_pointer_cast(keys_34.data());
insertKeys<<<std::ceil((float)COUNT34/128.), 128>>>(raw_keys34, d_HT_34.ref(cuco::insert), COUNT34);
int64_t* d_cycles_per_warp_main_21_aggregation_34;
auto main_21_aggregation_34_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_21_aggregation_34, sizeof(int64_t) * main_21_aggregation_34_cpw_size);
hipMemset(d_cycles_per_warp_main_21_aggregation_34, -1, sizeof(int64_t) * main_21_aggregation_34_cpw_size);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT34);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT34);
DBI32Type* d_KEY_34date__d_year;
hipMalloc(&d_KEY_34date__d_year, sizeof(DBI32Type) * COUNT34);
hipMemset(d_KEY_34date__d_year, 0, sizeof(DBI32Type) * COUNT34);
DBI16Type* d_KEY_34supplier__s_nation_encoded;
hipMalloc(&d_KEY_34supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT34);
hipMemset(d_KEY_34supplier__s_nation_encoded, 0, sizeof(DBI16Type) * COUNT34);
DBI16Type* d_KEY_34part__p_category_encoded;
hipMalloc(&d_KEY_34part__p_category_encoded, sizeof(DBI16Type) * COUNT34);
hipMemset(d_KEY_34part__p_category_encoded, 0, sizeof(DBI16Type) * COUNT34);
main_21<<<std::ceil((float)lineorder_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_29, d_BUF_30, d_BUF_31, d_BUF_32, d_HT_29.ref(cuco::find), d_HT_30.ref(cuco::find), d_HT_31.ref(cuco::find), d_HT_32.ref(cuco::find), d_HT_34.ref(cuco::find), d_KEY_34date__d_year, d_KEY_34part__p_category_encoded, d_KEY_34supplier__s_nation_encoded, d_aggr0__tmp_attr0, d_cycles_per_warp_main_21_aggregation_34, d_cycles_per_warp_main_21_join_probe_29, d_cycles_per_warp_main_21_join_probe_30, d_cycles_per_warp_main_21_join_probe_31, d_cycles_per_warp_main_21_join_probe_32, d_cycles_per_warp_main_21_map_33, d_cycles_per_warp_main_21_selection_20, d_cycles_per_warp_main_21_selection_22, d_cycles_per_warp_main_21_selection_23, d_cycles_per_warp_main_21_selection_24, d_cycles_per_warp_main_21_selection_25, d_cycles_per_warp_main_21_selection_26, d_cycles_per_warp_main_21_selection_27, d_cycles_per_warp_main_21_selection_28, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, d_lineorder__lo_supplycost, lineorder_size, d_part__p_category_encoded, d_supplier__s_nation_encoded);
int64_t* cycles_per_warp_main_21_selection_20 = (int64_t*)malloc(sizeof(int64_t) * main_21_selection_20_cpw_size);
hipMemcpy(cycles_per_warp_main_21_selection_20, d_cycles_per_warp_main_21_selection_20, sizeof(int64_t) * main_21_selection_20_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_21_selection_20 ";
for (auto i=0ull; i < main_21_selection_20_cpw_size; i++) std::cout << cycles_per_warp_main_21_selection_20[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_21_selection_22 = (int64_t*)malloc(sizeof(int64_t) * main_21_selection_22_cpw_size);
hipMemcpy(cycles_per_warp_main_21_selection_22, d_cycles_per_warp_main_21_selection_22, sizeof(int64_t) * main_21_selection_22_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_21_selection_22 ";
for (auto i=0ull; i < main_21_selection_22_cpw_size; i++) std::cout << cycles_per_warp_main_21_selection_22[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_21_selection_23 = (int64_t*)malloc(sizeof(int64_t) * main_21_selection_23_cpw_size);
hipMemcpy(cycles_per_warp_main_21_selection_23, d_cycles_per_warp_main_21_selection_23, sizeof(int64_t) * main_21_selection_23_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_21_selection_23 ";
for (auto i=0ull; i < main_21_selection_23_cpw_size; i++) std::cout << cycles_per_warp_main_21_selection_23[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_21_selection_24 = (int64_t*)malloc(sizeof(int64_t) * main_21_selection_24_cpw_size);
hipMemcpy(cycles_per_warp_main_21_selection_24, d_cycles_per_warp_main_21_selection_24, sizeof(int64_t) * main_21_selection_24_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_21_selection_24 ";
for (auto i=0ull; i < main_21_selection_24_cpw_size; i++) std::cout << cycles_per_warp_main_21_selection_24[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_21_selection_25 = (int64_t*)malloc(sizeof(int64_t) * main_21_selection_25_cpw_size);
hipMemcpy(cycles_per_warp_main_21_selection_25, d_cycles_per_warp_main_21_selection_25, sizeof(int64_t) * main_21_selection_25_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_21_selection_25 ";
for (auto i=0ull; i < main_21_selection_25_cpw_size; i++) std::cout << cycles_per_warp_main_21_selection_25[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_21_selection_26 = (int64_t*)malloc(sizeof(int64_t) * main_21_selection_26_cpw_size);
hipMemcpy(cycles_per_warp_main_21_selection_26, d_cycles_per_warp_main_21_selection_26, sizeof(int64_t) * main_21_selection_26_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_21_selection_26 ";
for (auto i=0ull; i < main_21_selection_26_cpw_size; i++) std::cout << cycles_per_warp_main_21_selection_26[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_21_selection_27 = (int64_t*)malloc(sizeof(int64_t) * main_21_selection_27_cpw_size);
hipMemcpy(cycles_per_warp_main_21_selection_27, d_cycles_per_warp_main_21_selection_27, sizeof(int64_t) * main_21_selection_27_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_21_selection_27 ";
for (auto i=0ull; i < main_21_selection_27_cpw_size; i++) std::cout << cycles_per_warp_main_21_selection_27[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_21_selection_28 = (int64_t*)malloc(sizeof(int64_t) * main_21_selection_28_cpw_size);
hipMemcpy(cycles_per_warp_main_21_selection_28, d_cycles_per_warp_main_21_selection_28, sizeof(int64_t) * main_21_selection_28_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_21_selection_28 ";
for (auto i=0ull; i < main_21_selection_28_cpw_size; i++) std::cout << cycles_per_warp_main_21_selection_28[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_21_join_probe_29 = (int64_t*)malloc(sizeof(int64_t) * main_21_join_probe_29_cpw_size);
hipMemcpy(cycles_per_warp_main_21_join_probe_29, d_cycles_per_warp_main_21_join_probe_29, sizeof(int64_t) * main_21_join_probe_29_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_21_join_probe_29 ";
for (auto i=0ull; i < main_21_join_probe_29_cpw_size; i++) std::cout << cycles_per_warp_main_21_join_probe_29[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_21_join_probe_30 = (int64_t*)malloc(sizeof(int64_t) * main_21_join_probe_30_cpw_size);
hipMemcpy(cycles_per_warp_main_21_join_probe_30, d_cycles_per_warp_main_21_join_probe_30, sizeof(int64_t) * main_21_join_probe_30_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_21_join_probe_30 ";
for (auto i=0ull; i < main_21_join_probe_30_cpw_size; i++) std::cout << cycles_per_warp_main_21_join_probe_30[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_21_join_probe_31 = (int64_t*)malloc(sizeof(int64_t) * main_21_join_probe_31_cpw_size);
hipMemcpy(cycles_per_warp_main_21_join_probe_31, d_cycles_per_warp_main_21_join_probe_31, sizeof(int64_t) * main_21_join_probe_31_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_21_join_probe_31 ";
for (auto i=0ull; i < main_21_join_probe_31_cpw_size; i++) std::cout << cycles_per_warp_main_21_join_probe_31[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_21_join_probe_32 = (int64_t*)malloc(sizeof(int64_t) * main_21_join_probe_32_cpw_size);
hipMemcpy(cycles_per_warp_main_21_join_probe_32, d_cycles_per_warp_main_21_join_probe_32, sizeof(int64_t) * main_21_join_probe_32_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_21_join_probe_32 ";
for (auto i=0ull; i < main_21_join_probe_32_cpw_size; i++) std::cout << cycles_per_warp_main_21_join_probe_32[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_21_map_33 = (int64_t*)malloc(sizeof(int64_t) * main_21_map_33_cpw_size);
hipMemcpy(cycles_per_warp_main_21_map_33, d_cycles_per_warp_main_21_map_33, sizeof(int64_t) * main_21_map_33_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_21_map_33 ";
for (auto i=0ull; i < main_21_map_33_cpw_size; i++) std::cout << cycles_per_warp_main_21_map_33[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_21_aggregation_34 = (int64_t*)malloc(sizeof(int64_t) * main_21_aggregation_34_cpw_size);
hipMemcpy(cycles_per_warp_main_21_aggregation_34, d_cycles_per_warp_main_21_aggregation_34, sizeof(int64_t) * main_21_aggregation_34_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_21_aggregation_34 ";
for (auto i=0ull; i < main_21_aggregation_34_cpw_size; i++) std::cout << cycles_per_warp_main_21_aggregation_34[i] << " ";
std::cout << std::endl;
//Materialize count
uint64_t* d_COUNT35;
hipMalloc(&d_COUNT35, sizeof(uint64_t));
hipMemset(d_COUNT35, 0, sizeof(uint64_t));
count_36<<<std::ceil((float)COUNT34/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT34, d_COUNT35);
uint64_t COUNT35;
hipMemcpy(&COUNT35, d_COUNT35, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_36_materialize_35;
auto main_36_materialize_35_cpw_size = std::ceil((float)COUNT34/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_36_materialize_35, sizeof(int64_t) * main_36_materialize_35_cpw_size);
hipMemset(d_cycles_per_warp_main_36_materialize_35, -1, sizeof(int64_t) * main_36_materialize_35_cpw_size);
//Materialize buffers
uint64_t* d_MAT_IDX35;
hipMalloc(&d_MAT_IDX35, sizeof(uint64_t));
hipMemset(d_MAT_IDX35, 0, sizeof(uint64_t));
auto MAT35date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT35);
DBI32Type* d_MAT35date__d_year;
hipMalloc(&d_MAT35date__d_year, sizeof(DBI32Type) * COUNT35);
auto MAT35supplier__s_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT35);
DBI16Type* d_MAT35supplier__s_nation_encoded;
hipMalloc(&d_MAT35supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT35);
auto MAT35part__p_category_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT35);
DBI16Type* d_MAT35part__p_category_encoded;
hipMalloc(&d_MAT35part__p_category_encoded, sizeof(DBI16Type) * COUNT35);
auto MAT35aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT35);
DBDecimalType* d_MAT35aggr0__tmp_attr0;
hipMalloc(&d_MAT35aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT35);
main_36<<<std::ceil((float)COUNT34/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT34, d_MAT35aggr0__tmp_attr0, d_MAT35date__d_year, d_MAT35part__p_category_encoded, d_MAT35supplier__s_nation_encoded, d_MAT_IDX35, d_aggr0__tmp_attr0, d_cycles_per_warp_main_36_materialize_35, d_KEY_34date__d_year, d_KEY_34part__p_category_encoded, d_KEY_34supplier__s_nation_encoded);
hipMemcpy(MAT35date__d_year, d_MAT35date__d_year, sizeof(DBI32Type) * COUNT35, hipMemcpyDeviceToHost);
hipMemcpy(MAT35supplier__s_nation_encoded, d_MAT35supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT35, hipMemcpyDeviceToHost);
hipMemcpy(MAT35part__p_category_encoded, d_MAT35part__p_category_encoded, sizeof(DBI16Type) * COUNT35, hipMemcpyDeviceToHost);
hipMemcpy(MAT35aggr0__tmp_attr0, d_MAT35aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT35, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_36_materialize_35 = (int64_t*)malloc(sizeof(int64_t) * main_36_materialize_35_cpw_size);
hipMemcpy(cycles_per_warp_main_36_materialize_35, d_cycles_per_warp_main_36_materialize_35, sizeof(int64_t) * main_36_materialize_35_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_36_materialize_35 ";
for (auto i=0ull; i < main_36_materialize_35_cpw_size; i++) std::cout << cycles_per_warp_main_36_materialize_35[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
      size_t aux_mem = usedGpuMem() - used_mem;
      std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_29);
hipFree(d_BUF_IDX_29);
hipFree(d_COUNT29);
hipFree(d_BUF_30);
hipFree(d_BUF_IDX_30);
hipFree(d_COUNT30);
hipFree(d_BUF_31);
hipFree(d_BUF_IDX_31);
hipFree(d_COUNT31);
hipFree(d_BUF_32);
hipFree(d_BUF_IDX_32);
hipFree(d_COUNT32);
hipFree(d_KEY_34date__d_year);
hipFree(d_KEY_34part__p_category_encoded);
hipFree(d_KEY_34supplier__s_nation_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT35);
hipFree(d_MAT35aggr0__tmp_attr0);
hipFree(d_MAT35date__d_year);
hipFree(d_MAT35part__p_category_encoded);
hipFree(d_MAT35supplier__s_nation_encoded);
hipFree(d_MAT_IDX35);
free(MAT35aggr0__tmp_attr0);
free(MAT35date__d_year);
free(MAT35part__p_category_encoded);
free(MAT35supplier__s_nation_encoded);
}