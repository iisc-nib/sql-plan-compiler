#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_57a6dbe5ad00(uint64_t* COUNT57a6dbe4cf70, DBStringType* part__p_brand1, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_brand1 = part__p_brand1[tid];
if (!(evaluatePredicate(reg_part__p_brand1, "MFGR#2239", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT57a6dbe4cf70, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_57a6dbe5ad00(uint64_t* BUF_57a6dbe4cf70, uint64_t* BUF_IDX_57a6dbe4cf70, HASHTABLE_INSERT HT_57a6dbe4cf70, DBStringType* part__p_brand1, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_brand1 = part__p_brand1[tid];
if (!(evaluatePredicate(reg_part__p_brand1, "MFGR#2239", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_57a6dbe4cf70 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_57a6dbe4cf70 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_57a6dbe4cf70 = atomicAdd((int*)BUF_IDX_57a6dbe4cf70, 1);
HT_57a6dbe4cf70.insert(cuco::pair{KEY_57a6dbe4cf70, buf_idx_57a6dbe4cf70});
BUF_57a6dbe4cf70[buf_idx_57a6dbe4cf70 * 1 + 0] = tid;
}
__global__ void count_57a6dbe5d720(uint64_t* COUNT57a6dbe49bc0, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "EUROPE", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT57a6dbe49bc0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_57a6dbe5d720(uint64_t* BUF_57a6dbe49bc0, uint64_t* BUF_IDX_57a6dbe49bc0, HASHTABLE_INSERT HT_57a6dbe49bc0, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "EUROPE", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_57a6dbe49bc0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_57a6dbe49bc0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_57a6dbe49bc0 = atomicAdd((int*)BUF_IDX_57a6dbe49bc0, 1);
HT_57a6dbe49bc0.insert(cuco::pair{KEY_57a6dbe49bc0, buf_idx_57a6dbe49bc0});
BUF_57a6dbe49bc0[buf_idx_57a6dbe49bc0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_57a6dbe32d80(uint64_t* BUF_57a6dbe49bc0, uint64_t* BUF_57a6dbe4cf70, uint64_t* COUNT57a6dbe4fb00, HASHTABLE_PROBE HT_57a6dbe49bc0, HASHTABLE_PROBE HT_57a6dbe4cf70, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_57a6dbe4cf70 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_57a6dbe4cf70 |= reg_lineorder__lo_partkey;
//Probe Hash table
auto SLOT_57a6dbe4cf70 = HT_57a6dbe4cf70.find(KEY_57a6dbe4cf70);
if (SLOT_57a6dbe4cf70 == HT_57a6dbe4cf70.end()) return;
if (!(true)) return;
uint64_t KEY_57a6dbe49bc0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_57a6dbe49bc0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_57a6dbe49bc0 = HT_57a6dbe49bc0.find(KEY_57a6dbe49bc0);
if (SLOT_57a6dbe49bc0 == HT_57a6dbe49bc0.end()) return;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT57a6dbe4fb00, 1);
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_57a6dbe32d80(uint64_t* BUF_57a6dbe49bc0, uint64_t* BUF_57a6dbe4cf70, uint64_t* BUF_57a6dbe4fb00, uint64_t* BUF_IDX_57a6dbe4fb00, HASHTABLE_PROBE HT_57a6dbe49bc0, HASHTABLE_PROBE HT_57a6dbe4cf70, HASHTABLE_INSERT HT_57a6dbe4fb00, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_57a6dbe4cf70 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_57a6dbe4cf70 |= reg_lineorder__lo_partkey;
//Probe Hash table
auto SLOT_57a6dbe4cf70 = HT_57a6dbe4cf70.find(KEY_57a6dbe4cf70);
if (SLOT_57a6dbe4cf70 == HT_57a6dbe4cf70.end()) return;
if (!(true)) return;
uint64_t KEY_57a6dbe49bc0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_57a6dbe49bc0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_57a6dbe49bc0 = HT_57a6dbe49bc0.find(KEY_57a6dbe49bc0);
if (SLOT_57a6dbe49bc0 == HT_57a6dbe49bc0.end()) return;
if (!(true)) return;
uint64_t KEY_57a6dbe4fb00 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_57a6dbe4fb00 |= reg_lineorder__lo_orderdate;
// Insert hash table kernel;
auto buf_idx_57a6dbe4fb00 = atomicAdd((int*)BUF_IDX_57a6dbe4fb00, 1);
HT_57a6dbe4fb00.insert(cuco::pair{KEY_57a6dbe4fb00, buf_idx_57a6dbe4fb00});
BUF_57a6dbe4fb00[buf_idx_57a6dbe4fb00 * 3 + 0] = tid;
BUF_57a6dbe4fb00[buf_idx_57a6dbe4fb00 * 3 + 1] = BUF_57a6dbe4cf70[SLOT_57a6dbe4cf70->second * 1 + 0];
BUF_57a6dbe4fb00[buf_idx_57a6dbe4fb00 * 3 + 2] = BUF_57a6dbe49bc0[SLOT_57a6dbe49bc0->second * 1 + 0];
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_57a6dbe33350(uint64_t* BUF_57a6dbe4fb00, HASHTABLE_INSERT HT_57a6dbe03310, HASHTABLE_PROBE HT_57a6dbe4fb00, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_57a6dbe4fb00 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_57a6dbe4fb00 |= reg_date__d_datekey;
//Probe Hash table
auto SLOT_57a6dbe4fb00 = HT_57a6dbe4fb00.find(KEY_57a6dbe4fb00);
if (SLOT_57a6dbe4fb00 == HT_57a6dbe4fb00.end()) return;
if (!(true)) return;
uint64_t KEY_57a6dbe03310 = 0;
auto reg_date__d_year = date__d_year[tid];

KEY_57a6dbe03310 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_57a6dbe4fb00[SLOT_57a6dbe4fb00->second * 3 + 1]];
KEY_57a6dbe03310 <<= 16;
KEY_57a6dbe03310 |= reg_part__p_brand1_encoded;
//Create aggregation hash table
HT_57a6dbe03310.insert(cuco::pair{KEY_57a6dbe03310, 1});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_57a6dbe33350(uint64_t* BUF_57a6dbe4fb00, HASHTABLE_FIND HT_57a6dbe03310, HASHTABLE_PROBE HT_57a6dbe4fb00, DBI32Type* KEY_57a6dbe03310date__d_year, DBI16Type* KEY_57a6dbe03310part__p_brand1_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size, DBDecimalType* lineorder__lo_revenue, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_57a6dbe4fb00 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_57a6dbe4fb00 |= reg_date__d_datekey;
//Probe Hash table
auto SLOT_57a6dbe4fb00 = HT_57a6dbe4fb00.find(KEY_57a6dbe4fb00);
if (SLOT_57a6dbe4fb00 == HT_57a6dbe4fb00.end()) return;
if (!(true)) return;
uint64_t KEY_57a6dbe03310 = 0;
auto reg_date__d_year = date__d_year[tid];

KEY_57a6dbe03310 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_57a6dbe4fb00[SLOT_57a6dbe4fb00->second * 3 + 1]];
KEY_57a6dbe03310 <<= 16;
KEY_57a6dbe03310 |= reg_part__p_brand1_encoded;
//Aggregate in hashtable
auto buf_idx_57a6dbe03310 = HT_57a6dbe03310.find(KEY_57a6dbe03310)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_57a6dbe4fb00[SLOT_57a6dbe4fb00->second * 3 + 0]];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_57a6dbe03310], reg_lineorder__lo_revenue);
KEY_57a6dbe03310date__d_year[buf_idx_57a6dbe03310] = reg_date__d_year;
KEY_57a6dbe03310part__p_brand1_encoded[buf_idx_57a6dbe03310] = reg_part__p_brand1_encoded;
}
__global__ void count_57a6dbe6e4d0(size_t COUNT57a6dbe03310, uint64_t* COUNT57a6dbe168b0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT57a6dbe03310) return;
//Materialize count
atomicAdd((int*)COUNT57a6dbe168b0, 1);
}
__global__ void main_57a6dbe6e4d0(size_t COUNT57a6dbe03310, DBDecimalType* MAT57a6dbe168b0aggr0__tmp_attr0, DBI32Type* MAT57a6dbe168b0date__d_year, DBI16Type* MAT57a6dbe168b0part__p_brand1_encoded, uint64_t* MAT_IDX57a6dbe168b0, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT57a6dbe03310) return;
//Materialize buffers
auto mat_idx57a6dbe168b0 = atomicAdd((int*)MAT_IDX57a6dbe168b0, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT57a6dbe168b0aggr0__tmp_attr0[mat_idx57a6dbe168b0] = reg_aggr0__tmp_attr0;
auto reg_date__d_year = date__d_year[tid];
MAT57a6dbe168b0date__d_year[mat_idx57a6dbe168b0] = reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[tid];
MAT57a6dbe168b0part__p_brand1_encoded[mat_idx57a6dbe168b0] = reg_part__p_brand1_encoded;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT57a6dbe4cf70;
hipMalloc(&d_COUNT57a6dbe4cf70, sizeof(uint64_t));
hipMemset(d_COUNT57a6dbe4cf70, 0, sizeof(uint64_t));
count_57a6dbe5ad00<<<std::ceil((float)part_size/128.), 128>>>(d_COUNT57a6dbe4cf70, d_part__p_brand1, part_size);
uint64_t COUNT57a6dbe4cf70;
hipMemcpy(&COUNT57a6dbe4cf70, d_COUNT57a6dbe4cf70, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_57a6dbe4cf70;
hipMalloc(&d_BUF_IDX_57a6dbe4cf70, sizeof(uint64_t));
hipMemset(d_BUF_IDX_57a6dbe4cf70, 0, sizeof(uint64_t));
uint64_t* d_BUF_57a6dbe4cf70;
hipMalloc(&d_BUF_57a6dbe4cf70, sizeof(uint64_t) * COUNT57a6dbe4cf70 * 1);
auto d_HT_57a6dbe4cf70 = cuco::static_map{ (int)COUNT57a6dbe4cf70*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_57a6dbe5ad00<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_57a6dbe4cf70, d_BUF_IDX_57a6dbe4cf70, d_HT_57a6dbe4cf70.ref(cuco::insert), d_part__p_brand1, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT57a6dbe49bc0;
hipMalloc(&d_COUNT57a6dbe49bc0, sizeof(uint64_t));
hipMemset(d_COUNT57a6dbe49bc0, 0, sizeof(uint64_t));
count_57a6dbe5d720<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT57a6dbe49bc0, d_supplier__s_region, supplier_size);
uint64_t COUNT57a6dbe49bc0;
hipMemcpy(&COUNT57a6dbe49bc0, d_COUNT57a6dbe49bc0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_57a6dbe49bc0;
hipMalloc(&d_BUF_IDX_57a6dbe49bc0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_57a6dbe49bc0, 0, sizeof(uint64_t));
uint64_t* d_BUF_57a6dbe49bc0;
hipMalloc(&d_BUF_57a6dbe49bc0, sizeof(uint64_t) * COUNT57a6dbe49bc0 * 1);
auto d_HT_57a6dbe49bc0 = cuco::static_map{ (int)COUNT57a6dbe49bc0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_57a6dbe5d720<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_57a6dbe49bc0, d_BUF_IDX_57a6dbe49bc0, d_HT_57a6dbe49bc0.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT57a6dbe4fb00;
hipMalloc(&d_COUNT57a6dbe4fb00, sizeof(uint64_t));
hipMemset(d_COUNT57a6dbe4fb00, 0, sizeof(uint64_t));
count_57a6dbe32d80<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_57a6dbe49bc0, d_BUF_57a6dbe4cf70, d_COUNT57a6dbe4fb00, d_HT_57a6dbe49bc0.ref(cuco::find), d_HT_57a6dbe4cf70.ref(cuco::find), d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
uint64_t COUNT57a6dbe4fb00;
hipMemcpy(&COUNT57a6dbe4fb00, d_COUNT57a6dbe4fb00, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_57a6dbe4fb00;
hipMalloc(&d_BUF_IDX_57a6dbe4fb00, sizeof(uint64_t));
hipMemset(d_BUF_IDX_57a6dbe4fb00, 0, sizeof(uint64_t));
uint64_t* d_BUF_57a6dbe4fb00;
hipMalloc(&d_BUF_57a6dbe4fb00, sizeof(uint64_t) * COUNT57a6dbe4fb00 * 3);
auto d_HT_57a6dbe4fb00 = cuco::static_map{ (int)COUNT57a6dbe4fb00*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_57a6dbe32d80<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_57a6dbe49bc0, d_BUF_57a6dbe4cf70, d_BUF_57a6dbe4fb00, d_BUF_IDX_57a6dbe4fb00, d_HT_57a6dbe49bc0.ref(cuco::find), d_HT_57a6dbe4cf70.ref(cuco::find), d_HT_57a6dbe4fb00.ref(cuco::insert), d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
//Create aggregation hash table
auto d_HT_57a6dbe03310 = cuco::static_map{ (int)1208*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_57a6dbe33350<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_57a6dbe4fb00, d_HT_57a6dbe03310.ref(cuco::insert), d_HT_57a6dbe4fb00.ref(cuco::find), d_date__d_datekey, d_date__d_year, date_size, d_part__p_brand1_encoded);
size_t COUNT57a6dbe03310 = d_HT_57a6dbe03310.size();
thrust::device_vector<int64_t> keys_57a6dbe03310(COUNT57a6dbe03310), vals_57a6dbe03310(COUNT57a6dbe03310);
d_HT_57a6dbe03310.retrieve_all(keys_57a6dbe03310.begin(), vals_57a6dbe03310.begin());
d_HT_57a6dbe03310.clear();
int64_t* raw_keys57a6dbe03310 = thrust::raw_pointer_cast(keys_57a6dbe03310.data());
insertKeys<<<std::ceil((float)COUNT57a6dbe03310/128.), 128>>>(raw_keys57a6dbe03310, d_HT_57a6dbe03310.ref(cuco::insert), COUNT57a6dbe03310);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT57a6dbe03310);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT57a6dbe03310);
DBI32Type* d_KEY_57a6dbe03310date__d_year;
hipMalloc(&d_KEY_57a6dbe03310date__d_year, sizeof(DBI32Type) * COUNT57a6dbe03310);
hipMemset(d_KEY_57a6dbe03310date__d_year, 0, sizeof(DBI32Type) * COUNT57a6dbe03310);
DBI16Type* d_KEY_57a6dbe03310part__p_brand1_encoded;
hipMalloc(&d_KEY_57a6dbe03310part__p_brand1_encoded, sizeof(DBI16Type) * COUNT57a6dbe03310);
hipMemset(d_KEY_57a6dbe03310part__p_brand1_encoded, 0, sizeof(DBI16Type) * COUNT57a6dbe03310);
main_57a6dbe33350<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_57a6dbe4fb00, d_HT_57a6dbe03310.ref(cuco::find), d_HT_57a6dbe4fb00.ref(cuco::find), d_KEY_57a6dbe03310date__d_year, d_KEY_57a6dbe03310part__p_brand1_encoded, d_aggr0__tmp_attr0, d_date__d_datekey, d_date__d_year, date_size, d_lineorder__lo_revenue, d_part__p_brand1_encoded);
//Materialize count
uint64_t* d_COUNT57a6dbe168b0;
hipMalloc(&d_COUNT57a6dbe168b0, sizeof(uint64_t));
hipMemset(d_COUNT57a6dbe168b0, 0, sizeof(uint64_t));
count_57a6dbe6e4d0<<<std::ceil((float)COUNT57a6dbe03310/128.), 128>>>(COUNT57a6dbe03310, d_COUNT57a6dbe168b0);
uint64_t COUNT57a6dbe168b0;
hipMemcpy(&COUNT57a6dbe168b0, d_COUNT57a6dbe168b0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX57a6dbe168b0;
hipMalloc(&d_MAT_IDX57a6dbe168b0, sizeof(uint64_t));
hipMemset(d_MAT_IDX57a6dbe168b0, 0, sizeof(uint64_t));
auto MAT57a6dbe168b0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT57a6dbe168b0);
DBDecimalType* d_MAT57a6dbe168b0aggr0__tmp_attr0;
hipMalloc(&d_MAT57a6dbe168b0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT57a6dbe168b0);
auto MAT57a6dbe168b0date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT57a6dbe168b0);
DBI32Type* d_MAT57a6dbe168b0date__d_year;
hipMalloc(&d_MAT57a6dbe168b0date__d_year, sizeof(DBI32Type) * COUNT57a6dbe168b0);
auto MAT57a6dbe168b0part__p_brand1_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT57a6dbe168b0);
DBI16Type* d_MAT57a6dbe168b0part__p_brand1_encoded;
hipMalloc(&d_MAT57a6dbe168b0part__p_brand1_encoded, sizeof(DBI16Type) * COUNT57a6dbe168b0);
main_57a6dbe6e4d0<<<std::ceil((float)COUNT57a6dbe03310/128.), 128>>>(COUNT57a6dbe03310, d_MAT57a6dbe168b0aggr0__tmp_attr0, d_MAT57a6dbe168b0date__d_year, d_MAT57a6dbe168b0part__p_brand1_encoded, d_MAT_IDX57a6dbe168b0, d_aggr0__tmp_attr0, d_KEY_57a6dbe03310date__d_year, d_KEY_57a6dbe03310part__p_brand1_encoded);
hipMemcpy(MAT57a6dbe168b0aggr0__tmp_attr0, d_MAT57a6dbe168b0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT57a6dbe168b0, hipMemcpyDeviceToHost);
hipMemcpy(MAT57a6dbe168b0date__d_year, d_MAT57a6dbe168b0date__d_year, sizeof(DBI32Type) * COUNT57a6dbe168b0, hipMemcpyDeviceToHost);
hipMemcpy(MAT57a6dbe168b0part__p_brand1_encoded, d_MAT57a6dbe168b0part__p_brand1_encoded, sizeof(DBI16Type) * COUNT57a6dbe168b0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT57a6dbe168b0; i++) { std::cout << "" << MAT57a6dbe168b0aggr0__tmp_attr0[i];
std::cout << "," << MAT57a6dbe168b0date__d_year[i];
std::cout << "," << part__p_brand1_map[MAT57a6dbe168b0part__p_brand1_encoded[i]];
std::cout << std::endl; }
hipFree(d_BUF_57a6dbe4cf70);
hipFree(d_BUF_IDX_57a6dbe4cf70);
hipFree(d_COUNT57a6dbe4cf70);
hipFree(d_BUF_57a6dbe49bc0);
hipFree(d_BUF_IDX_57a6dbe49bc0);
hipFree(d_COUNT57a6dbe49bc0);
hipFree(d_BUF_57a6dbe4fb00);
hipFree(d_BUF_IDX_57a6dbe4fb00);
hipFree(d_COUNT57a6dbe4fb00);
hipFree(d_KEY_57a6dbe03310date__d_year);
hipFree(d_KEY_57a6dbe03310part__p_brand1_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT57a6dbe168b0);
hipFree(d_MAT57a6dbe168b0aggr0__tmp_attr0);
hipFree(d_MAT57a6dbe168b0date__d_year);
hipFree(d_MAT57a6dbe168b0part__p_brand1_encoded);
hipFree(d_MAT_IDX57a6dbe168b0);
free(MAT57a6dbe168b0aggr0__tmp_attr0);
free(MAT57a6dbe168b0date__d_year);
free(MAT57a6dbe168b0part__p_brand1_encoded);
}