#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_55cb94e8fff0(uint64_t* COUNT55cb94e85850, DBStringType* part__p_brand1, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_brand1 = part__p_brand1[tid];
if (!(evaluatePredicate(reg_part__p_brand1, "MFGR#2221", Predicate::gte) && evaluatePredicate(reg_part__p_brand1, "MFGR#2228", Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT55cb94e85850, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_55cb94e8fff0(uint64_t* BUF_55cb94e85850, uint64_t* BUF_IDX_55cb94e85850, HASHTABLE_INSERT HT_55cb94e85850, DBStringType* part__p_brand1, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_brand1 = part__p_brand1[tid];
if (!(evaluatePredicate(reg_part__p_brand1, "MFGR#2221", Predicate::gte) && evaluatePredicate(reg_part__p_brand1, "MFGR#2228", Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_55cb94e85850 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_55cb94e85850 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_55cb94e85850 = atomicAdd((int*)BUF_IDX_55cb94e85850, 1);
HT_55cb94e85850.insert(cuco::pair{KEY_55cb94e85850, buf_idx_55cb94e85850});
BUF_55cb94e85850[buf_idx_55cb94e85850 * 1 + 0] = tid;
}
__global__ void count_55cb94e928f0(uint64_t* COUNT55cb94e85910, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT55cb94e85910, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_55cb94e928f0(uint64_t* BUF_55cb94e85910, uint64_t* BUF_IDX_55cb94e85910, HASHTABLE_INSERT HT_55cb94e85910, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_55cb94e85910 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_55cb94e85910 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_55cb94e85910 = atomicAdd((int*)BUF_IDX_55cb94e85910, 1);
HT_55cb94e85910.insert(cuco::pair{KEY_55cb94e85910, buf_idx_55cb94e85910});
BUF_55cb94e85910[buf_idx_55cb94e85910 * 1 + 0] = tid;
}
__global__ void count_55cb94e69db0(uint64_t* COUNT55cb94e80270, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT55cb94e80270, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_55cb94e69db0(uint64_t* BUF_55cb94e80270, uint64_t* BUF_IDX_55cb94e80270, HASHTABLE_INSERT HT_55cb94e80270, DBI32Type* date__d_datekey, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_55cb94e80270 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_55cb94e80270 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_55cb94e80270 = atomicAdd((int*)BUF_IDX_55cb94e80270, 1);
HT_55cb94e80270.insert(cuco::pair{KEY_55cb94e80270, buf_idx_55cb94e80270});
BUF_55cb94e80270[buf_idx_55cb94e80270 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_55cb94e697e0(uint64_t* BUF_55cb94e80270, uint64_t* BUF_55cb94e85850, uint64_t* BUF_55cb94e85910, HASHTABLE_INSERT HT_55cb94e3a380, HASHTABLE_PROBE HT_55cb94e80270, HASHTABLE_PROBE HT_55cb94e85850, HASHTABLE_PROBE HT_55cb94e85910, DBI32Type* date__d_year, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_55cb94e85850 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_55cb94e85850 |= reg_lineorder__lo_partkey;
//Probe Hash table
auto SLOT_55cb94e85850 = HT_55cb94e85850.find(KEY_55cb94e85850);
if (SLOT_55cb94e85850 == HT_55cb94e85850.end()) return;
if (!(true)) return;
uint64_t KEY_55cb94e85910 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_55cb94e85910 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_55cb94e85910 = HT_55cb94e85910.find(KEY_55cb94e85910);
if (SLOT_55cb94e85910 == HT_55cb94e85910.end()) return;
if (!(true)) return;
uint64_t KEY_55cb94e80270 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_55cb94e80270 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_55cb94e80270 = HT_55cb94e80270.find(KEY_55cb94e80270);
if (SLOT_55cb94e80270 == HT_55cb94e80270.end()) return;
if (!(true)) return;
uint64_t KEY_55cb94e3a380 = 0;
auto reg_date__d_year = date__d_year[BUF_55cb94e80270[SLOT_55cb94e80270->second * 1 + 0]];

KEY_55cb94e3a380 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_55cb94e85850[SLOT_55cb94e85850->second * 1 + 0]];
KEY_55cb94e3a380 <<= 16;
KEY_55cb94e3a380 |= reg_part__p_brand1_encoded;
//Create aggregation hash table
HT_55cb94e3a380.insert(cuco::pair{KEY_55cb94e3a380, 1});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_55cb94e697e0(uint64_t* BUF_55cb94e80270, uint64_t* BUF_55cb94e85850, uint64_t* BUF_55cb94e85910, HASHTABLE_FIND HT_55cb94e3a380, HASHTABLE_PROBE HT_55cb94e80270, HASHTABLE_PROBE HT_55cb94e85850, HASHTABLE_PROBE HT_55cb94e85910, DBI32Type* KEY_55cb94e3a380date__d_year, DBI16Type* KEY_55cb94e3a380part__p_brand1_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_55cb94e85850 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_55cb94e85850 |= reg_lineorder__lo_partkey;
//Probe Hash table
auto SLOT_55cb94e85850 = HT_55cb94e85850.find(KEY_55cb94e85850);
if (SLOT_55cb94e85850 == HT_55cb94e85850.end()) return;
if (!(true)) return;
uint64_t KEY_55cb94e85910 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_55cb94e85910 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_55cb94e85910 = HT_55cb94e85910.find(KEY_55cb94e85910);
if (SLOT_55cb94e85910 == HT_55cb94e85910.end()) return;
if (!(true)) return;
uint64_t KEY_55cb94e80270 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_55cb94e80270 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_55cb94e80270 = HT_55cb94e80270.find(KEY_55cb94e80270);
if (SLOT_55cb94e80270 == HT_55cb94e80270.end()) return;
if (!(true)) return;
uint64_t KEY_55cb94e3a380 = 0;
auto reg_date__d_year = date__d_year[BUF_55cb94e80270[SLOT_55cb94e80270->second * 1 + 0]];

KEY_55cb94e3a380 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_55cb94e85850[SLOT_55cb94e85850->second * 1 + 0]];
KEY_55cb94e3a380 <<= 16;
KEY_55cb94e3a380 |= reg_part__p_brand1_encoded;
//Aggregate in hashtable
auto buf_idx_55cb94e3a380 = HT_55cb94e3a380.find(KEY_55cb94e3a380)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_55cb94e3a380], reg_lineorder__lo_revenue);
KEY_55cb94e3a380date__d_year[buf_idx_55cb94e3a380] = reg_date__d_year;
KEY_55cb94e3a380part__p_brand1_encoded[buf_idx_55cb94e3a380] = reg_part__p_brand1_encoded;
}
__global__ void count_55cb94ea1e30(size_t COUNT55cb94e3a380, uint64_t* COUNT55cb94e4d3d0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT55cb94e3a380) return;
//Materialize count
atomicAdd((int*)COUNT55cb94e4d3d0, 1);
}
__global__ void main_55cb94ea1e30(size_t COUNT55cb94e3a380, DBDecimalType* MAT55cb94e4d3d0aggr0__tmp_attr0, DBI32Type* MAT55cb94e4d3d0date__d_year, DBI16Type* MAT55cb94e4d3d0part__p_brand1_encoded, uint64_t* MAT_IDX55cb94e4d3d0, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT55cb94e3a380) return;
//Materialize buffers
auto mat_idx55cb94e4d3d0 = atomicAdd((int*)MAT_IDX55cb94e4d3d0, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT55cb94e4d3d0aggr0__tmp_attr0[mat_idx55cb94e4d3d0] = reg_aggr0__tmp_attr0;
auto reg_date__d_year = date__d_year[tid];
MAT55cb94e4d3d0date__d_year[mat_idx55cb94e4d3d0] = reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[tid];
MAT55cb94e4d3d0part__p_brand1_encoded[mat_idx55cb94e4d3d0] = reg_part__p_brand1_encoded;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT55cb94e85850;
hipMalloc(&d_COUNT55cb94e85850, sizeof(uint64_t));
hipMemset(d_COUNT55cb94e85850, 0, sizeof(uint64_t));
count_55cb94e8fff0<<<std::ceil((float)part_size/128.), 128>>>(d_COUNT55cb94e85850, d_part__p_brand1, part_size);
uint64_t COUNT55cb94e85850;
hipMemcpy(&COUNT55cb94e85850, d_COUNT55cb94e85850, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_55cb94e85850;
hipMalloc(&d_BUF_IDX_55cb94e85850, sizeof(uint64_t));
hipMemset(d_BUF_IDX_55cb94e85850, 0, sizeof(uint64_t));
uint64_t* d_BUF_55cb94e85850;
hipMalloc(&d_BUF_55cb94e85850, sizeof(uint64_t) * COUNT55cb94e85850 * 1);
auto d_HT_55cb94e85850 = cuco::static_map{ (int)COUNT55cb94e85850*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_55cb94e8fff0<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_55cb94e85850, d_BUF_IDX_55cb94e85850, d_HT_55cb94e85850.ref(cuco::insert), d_part__p_brand1, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT55cb94e85910;
hipMalloc(&d_COUNT55cb94e85910, sizeof(uint64_t));
hipMemset(d_COUNT55cb94e85910, 0, sizeof(uint64_t));
count_55cb94e928f0<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT55cb94e85910, d_supplier__s_region, supplier_size);
uint64_t COUNT55cb94e85910;
hipMemcpy(&COUNT55cb94e85910, d_COUNT55cb94e85910, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_55cb94e85910;
hipMalloc(&d_BUF_IDX_55cb94e85910, sizeof(uint64_t));
hipMemset(d_BUF_IDX_55cb94e85910, 0, sizeof(uint64_t));
uint64_t* d_BUF_55cb94e85910;
hipMalloc(&d_BUF_55cb94e85910, sizeof(uint64_t) * COUNT55cb94e85910 * 1);
auto d_HT_55cb94e85910 = cuco::static_map{ (int)COUNT55cb94e85910*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_55cb94e928f0<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_55cb94e85910, d_BUF_IDX_55cb94e85910, d_HT_55cb94e85910.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT55cb94e80270;
hipMalloc(&d_COUNT55cb94e80270, sizeof(uint64_t));
hipMemset(d_COUNT55cb94e80270, 0, sizeof(uint64_t));
count_55cb94e69db0<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT55cb94e80270, date_size);
uint64_t COUNT55cb94e80270;
hipMemcpy(&COUNT55cb94e80270, d_COUNT55cb94e80270, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_55cb94e80270;
hipMalloc(&d_BUF_IDX_55cb94e80270, sizeof(uint64_t));
hipMemset(d_BUF_IDX_55cb94e80270, 0, sizeof(uint64_t));
uint64_t* d_BUF_55cb94e80270;
hipMalloc(&d_BUF_55cb94e80270, sizeof(uint64_t) * COUNT55cb94e80270 * 1);
auto d_HT_55cb94e80270 = cuco::static_map{ (int)COUNT55cb94e80270*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_55cb94e69db0<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_55cb94e80270, d_BUF_IDX_55cb94e80270, d_HT_55cb94e80270.ref(cuco::insert), d_date__d_datekey, date_size);
//Create aggregation hash table
auto d_HT_55cb94e3a380 = cuco::static_map{ (int)3846*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_55cb94e697e0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_55cb94e80270, d_BUF_55cb94e85850, d_BUF_55cb94e85910, d_HT_55cb94e3a380.ref(cuco::insert), d_HT_55cb94e80270.ref(cuco::find), d_HT_55cb94e85850.ref(cuco::find), d_HT_55cb94e85910.ref(cuco::find), d_date__d_year, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size, d_part__p_brand1_encoded);
size_t COUNT55cb94e3a380 = d_HT_55cb94e3a380.size();
thrust::device_vector<int64_t> keys_55cb94e3a380(COUNT55cb94e3a380), vals_55cb94e3a380(COUNT55cb94e3a380);
d_HT_55cb94e3a380.retrieve_all(keys_55cb94e3a380.begin(), vals_55cb94e3a380.begin());
d_HT_55cb94e3a380.clear();
int64_t* raw_keys55cb94e3a380 = thrust::raw_pointer_cast(keys_55cb94e3a380.data());
insertKeys<<<std::ceil((float)COUNT55cb94e3a380/128.), 128>>>(raw_keys55cb94e3a380, d_HT_55cb94e3a380.ref(cuco::insert), COUNT55cb94e3a380);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT55cb94e3a380);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT55cb94e3a380);
DBI32Type* d_KEY_55cb94e3a380date__d_year;
hipMalloc(&d_KEY_55cb94e3a380date__d_year, sizeof(DBI32Type) * COUNT55cb94e3a380);
hipMemset(d_KEY_55cb94e3a380date__d_year, 0, sizeof(DBI32Type) * COUNT55cb94e3a380);
DBI16Type* d_KEY_55cb94e3a380part__p_brand1_encoded;
hipMalloc(&d_KEY_55cb94e3a380part__p_brand1_encoded, sizeof(DBI16Type) * COUNT55cb94e3a380);
hipMemset(d_KEY_55cb94e3a380part__p_brand1_encoded, 0, sizeof(DBI16Type) * COUNT55cb94e3a380);
main_55cb94e697e0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_55cb94e80270, d_BUF_55cb94e85850, d_BUF_55cb94e85910, d_HT_55cb94e3a380.ref(cuco::find), d_HT_55cb94e80270.ref(cuco::find), d_HT_55cb94e85850.ref(cuco::find), d_HT_55cb94e85910.ref(cuco::find), d_KEY_55cb94e3a380date__d_year, d_KEY_55cb94e3a380part__p_brand1_encoded, d_aggr0__tmp_attr0, d_date__d_year, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_part__p_brand1_encoded);
//Materialize count
uint64_t* d_COUNT55cb94e4d3d0;
hipMalloc(&d_COUNT55cb94e4d3d0, sizeof(uint64_t));
hipMemset(d_COUNT55cb94e4d3d0, 0, sizeof(uint64_t));
count_55cb94ea1e30<<<std::ceil((float)COUNT55cb94e3a380/128.), 128>>>(COUNT55cb94e3a380, d_COUNT55cb94e4d3d0);
uint64_t COUNT55cb94e4d3d0;
hipMemcpy(&COUNT55cb94e4d3d0, d_COUNT55cb94e4d3d0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX55cb94e4d3d0;
hipMalloc(&d_MAT_IDX55cb94e4d3d0, sizeof(uint64_t));
hipMemset(d_MAT_IDX55cb94e4d3d0, 0, sizeof(uint64_t));
auto MAT55cb94e4d3d0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT55cb94e4d3d0);
DBDecimalType* d_MAT55cb94e4d3d0aggr0__tmp_attr0;
hipMalloc(&d_MAT55cb94e4d3d0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT55cb94e4d3d0);
auto MAT55cb94e4d3d0date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT55cb94e4d3d0);
DBI32Type* d_MAT55cb94e4d3d0date__d_year;
hipMalloc(&d_MAT55cb94e4d3d0date__d_year, sizeof(DBI32Type) * COUNT55cb94e4d3d0);
auto MAT55cb94e4d3d0part__p_brand1_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT55cb94e4d3d0);
DBI16Type* d_MAT55cb94e4d3d0part__p_brand1_encoded;
hipMalloc(&d_MAT55cb94e4d3d0part__p_brand1_encoded, sizeof(DBI16Type) * COUNT55cb94e4d3d0);
main_55cb94ea1e30<<<std::ceil((float)COUNT55cb94e3a380/128.), 128>>>(COUNT55cb94e3a380, d_MAT55cb94e4d3d0aggr0__tmp_attr0, d_MAT55cb94e4d3d0date__d_year, d_MAT55cb94e4d3d0part__p_brand1_encoded, d_MAT_IDX55cb94e4d3d0, d_aggr0__tmp_attr0, d_KEY_55cb94e3a380date__d_year, d_KEY_55cb94e3a380part__p_brand1_encoded);
hipMemcpy(MAT55cb94e4d3d0aggr0__tmp_attr0, d_MAT55cb94e4d3d0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT55cb94e4d3d0, hipMemcpyDeviceToHost);
hipMemcpy(MAT55cb94e4d3d0date__d_year, d_MAT55cb94e4d3d0date__d_year, sizeof(DBI32Type) * COUNT55cb94e4d3d0, hipMemcpyDeviceToHost);
hipMemcpy(MAT55cb94e4d3d0part__p_brand1_encoded, d_MAT55cb94e4d3d0part__p_brand1_encoded, sizeof(DBI16Type) * COUNT55cb94e4d3d0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT55cb94e4d3d0; i++) { std::cout << "" << MAT55cb94e4d3d0aggr0__tmp_attr0[i];
std::cout << "," << MAT55cb94e4d3d0date__d_year[i];
std::cout << "," << part__p_brand1_map[MAT55cb94e4d3d0part__p_brand1_encoded[i]];
std::cout << std::endl; }
hipFree(d_BUF_55cb94e85850);
hipFree(d_BUF_IDX_55cb94e85850);
hipFree(d_COUNT55cb94e85850);
hipFree(d_BUF_55cb94e85910);
hipFree(d_BUF_IDX_55cb94e85910);
hipFree(d_COUNT55cb94e85910);
hipFree(d_BUF_55cb94e80270);
hipFree(d_BUF_IDX_55cb94e80270);
hipFree(d_COUNT55cb94e80270);
hipFree(d_KEY_55cb94e3a380date__d_year);
hipFree(d_KEY_55cb94e3a380part__p_brand1_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT55cb94e4d3d0);
hipFree(d_MAT55cb94e4d3d0aggr0__tmp_attr0);
hipFree(d_MAT55cb94e4d3d0date__d_year);
hipFree(d_MAT55cb94e4d3d0part__p_brand1_encoded);
hipFree(d_MAT_IDX55cb94e4d3d0);
free(MAT55cb94e4d3d0aggr0__tmp_attr0);
free(MAT55cb94e4d3d0date__d_year);
free(MAT55cb94e4d3d0part__p_brand1_encoded);
}