#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_58a5f8f2f1e0(uint64_t* COUNT58a5f8f43da0, DBI32Type* date__d_yearmonthnum, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_yearmonthnum = date__d_yearmonthnum[tid];
if (!(evaluatePredicate(reg_date__d_yearmonthnum, 199401, Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT58a5f8f43da0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_58a5f8f2f1e0(uint64_t* BUF_58a5f8f43da0, uint64_t* BUF_IDX_58a5f8f43da0, HASHTABLE_INSERT HT_58a5f8f43da0, DBI32Type* date__d_datekey, DBI32Type* date__d_yearmonthnum, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_yearmonthnum = date__d_yearmonthnum[tid];
if (!(evaluatePredicate(reg_date__d_yearmonthnum, 199401, Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_58a5f8f43da0 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_58a5f8f43da0 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_58a5f8f43da0 = atomicAdd((int*)BUF_IDX_58a5f8f43da0, 1);
HT_58a5f8f43da0.insert(cuco::pair{KEY_58a5f8f43da0, buf_idx_58a5f8f43da0});
BUF_58a5f8f43da0[buf_idx_58a5f8f43da0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_58a5f8f2ed30(uint64_t* BUF_58a5f8f43da0, HASHTABLE_INSERT HT_58a5f8f00550, HASHTABLE_PROBE HT_58a5f8f43da0, DBI32Type* lineorder__lo_discount, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_discount = lineorder__lo_discount[tid];
if (!(evaluatePredicate(reg_lineorder__lo_discount, 4, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount, 6, Predicate::lte))) return;
auto reg_lineorder__lo_quantity = lineorder__lo_quantity[tid];
if (!(evaluatePredicate(reg_lineorder__lo_quantity, 26, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_quantity, 35, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_58a5f8f43da0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_58a5f8f43da0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_58a5f8f43da0 = HT_58a5f8f43da0.find(KEY_58a5f8f43da0);
if (SLOT_58a5f8f43da0 == HT_58a5f8f43da0.end()) return;
if (!(true)) return;
uint64_t KEY_58a5f8f00550 = 0;
//Create aggregation hash table
HT_58a5f8f00550.insert(cuco::pair{KEY_58a5f8f00550, 1});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_58a5f8f2ed30(uint64_t* BUF_58a5f8f43da0, HASHTABLE_FIND HT_58a5f8f00550, HASHTABLE_PROBE HT_58a5f8f43da0, DBDecimalType* aggr0__tmp_attr0, DBI32Type* lineorder__lo_discount, DBDecimalType* lineorder__lo_extendedprice, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_discount = lineorder__lo_discount[tid];
if (!(evaluatePredicate(reg_lineorder__lo_discount, 4, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount, 6, Predicate::lte))) return;
auto reg_lineorder__lo_quantity = lineorder__lo_quantity[tid];
if (!(evaluatePredicate(reg_lineorder__lo_quantity, 26, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_quantity, 35, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_58a5f8f43da0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_58a5f8f43da0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_58a5f8f43da0 = HT_58a5f8f43da0.find(KEY_58a5f8f43da0);
if (SLOT_58a5f8f43da0 == HT_58a5f8f43da0.end()) return;
if (!(true)) return;
uint64_t KEY_58a5f8f00550 = 0;
//Aggregate in hashtable
auto buf_idx_58a5f8f00550 = HT_58a5f8f00550.find(KEY_58a5f8f00550)->second;
auto reg_lineorder__lo_extendedprice = lineorder__lo_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_extendedprice) * ((DBDecimalType)(reg_lineorder__lo_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_58a5f8f00550], reg_map0__tmp_attr1);
}
__global__ void count_58a5f8f57ab0(size_t COUNT58a5f8f00550, uint64_t* COUNT58a5f8f12430) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT58a5f8f00550) return;
//Materialize count
atomicAdd((int*)COUNT58a5f8f12430, 1);
}
__global__ void main_58a5f8f57ab0(size_t COUNT58a5f8f00550, DBDecimalType* MAT58a5f8f12430aggr0__tmp_attr0, uint64_t* MAT_IDX58a5f8f12430, DBDecimalType* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT58a5f8f00550) return;
//Materialize buffers
auto mat_idx58a5f8f12430 = atomicAdd((int*)MAT_IDX58a5f8f12430, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT58a5f8f12430aggr0__tmp_attr0[mat_idx58a5f8f12430] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT58a5f8f43da0;
hipMalloc(&d_COUNT58a5f8f43da0, sizeof(uint64_t));
hipMemset(d_COUNT58a5f8f43da0, 0, sizeof(uint64_t));
count_58a5f8f2f1e0<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT58a5f8f43da0, d_date__d_yearmonthnum, date_size);
uint64_t COUNT58a5f8f43da0;
hipMemcpy(&COUNT58a5f8f43da0, d_COUNT58a5f8f43da0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_58a5f8f43da0;
hipMalloc(&d_BUF_IDX_58a5f8f43da0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_58a5f8f43da0, 0, sizeof(uint64_t));
uint64_t* d_BUF_58a5f8f43da0;
hipMalloc(&d_BUF_58a5f8f43da0, sizeof(uint64_t) * COUNT58a5f8f43da0 * 1);
auto d_HT_58a5f8f43da0 = cuco::static_map{ (int)COUNT58a5f8f43da0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_58a5f8f2f1e0<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_58a5f8f43da0, d_BUF_IDX_58a5f8f43da0, d_HT_58a5f8f43da0.ref(cuco::insert), d_date__d_datekey, d_date__d_yearmonthnum, date_size);
//Create aggregation hash table
auto d_HT_58a5f8f00550 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_58a5f8f2ed30<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_58a5f8f43da0, d_HT_58a5f8f00550.ref(cuco::insert), d_HT_58a5f8f43da0.ref(cuco::find), d_lineorder__lo_discount, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
size_t COUNT58a5f8f00550 = d_HT_58a5f8f00550.size();
thrust::device_vector<int64_t> keys_58a5f8f00550(COUNT58a5f8f00550), vals_58a5f8f00550(COUNT58a5f8f00550);
d_HT_58a5f8f00550.retrieve_all(keys_58a5f8f00550.begin(), vals_58a5f8f00550.begin());
d_HT_58a5f8f00550.clear();
int64_t* raw_keys58a5f8f00550 = thrust::raw_pointer_cast(keys_58a5f8f00550.data());
insertKeys<<<std::ceil((float)COUNT58a5f8f00550/128.), 128>>>(raw_keys58a5f8f00550, d_HT_58a5f8f00550.ref(cuco::insert), COUNT58a5f8f00550);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT58a5f8f00550);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT58a5f8f00550);
main_58a5f8f2ed30<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_58a5f8f43da0, d_HT_58a5f8f00550.ref(cuco::find), d_HT_58a5f8f43da0.ref(cuco::find), d_aggr0__tmp_attr0, d_lineorder__lo_discount, d_lineorder__lo_extendedprice, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
//Materialize count
uint64_t* d_COUNT58a5f8f12430;
hipMalloc(&d_COUNT58a5f8f12430, sizeof(uint64_t));
hipMemset(d_COUNT58a5f8f12430, 0, sizeof(uint64_t));
count_58a5f8f57ab0<<<std::ceil((float)COUNT58a5f8f00550/128.), 128>>>(COUNT58a5f8f00550, d_COUNT58a5f8f12430);
uint64_t COUNT58a5f8f12430;
hipMemcpy(&COUNT58a5f8f12430, d_COUNT58a5f8f12430, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX58a5f8f12430;
hipMalloc(&d_MAT_IDX58a5f8f12430, sizeof(uint64_t));
hipMemset(d_MAT_IDX58a5f8f12430, 0, sizeof(uint64_t));
auto MAT58a5f8f12430aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT58a5f8f12430);
DBDecimalType* d_MAT58a5f8f12430aggr0__tmp_attr0;
hipMalloc(&d_MAT58a5f8f12430aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT58a5f8f12430);
main_58a5f8f57ab0<<<std::ceil((float)COUNT58a5f8f00550/128.), 128>>>(COUNT58a5f8f00550, d_MAT58a5f8f12430aggr0__tmp_attr0, d_MAT_IDX58a5f8f12430, d_aggr0__tmp_attr0);
hipMemcpy(MAT58a5f8f12430aggr0__tmp_attr0, d_MAT58a5f8f12430aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT58a5f8f12430, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT58a5f8f12430; i++) { std::cout << "" << MAT58a5f8f12430aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_58a5f8f43da0);
hipFree(d_BUF_IDX_58a5f8f43da0);
hipFree(d_COUNT58a5f8f43da0);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT58a5f8f12430);
hipFree(d_MAT58a5f8f12430aggr0__tmp_attr0);
hipFree(d_MAT_IDX58a5f8f12430);
free(MAT58a5f8f12430aggr0__tmp_attr0);
}