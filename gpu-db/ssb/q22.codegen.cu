#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_595111dcbfd0(uint64_t* COUNT595111dc24f0, DBStringType* part__p_brand1, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_brand1 = part__p_brand1[tid];
if (!(evaluatePredicate(reg_part__p_brand1, "MFGR#2221", Predicate::gte) && evaluatePredicate(reg_part__p_brand1, "MFGR#2228", Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT595111dc24f0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_595111dcbfd0(uint64_t* BUF_595111dc24f0, uint64_t* BUF_IDX_595111dc24f0, HASHTABLE_INSERT HT_595111dc24f0, DBStringType* part__p_brand1, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_brand1 = part__p_brand1[tid];
if (!(evaluatePredicate(reg_part__p_brand1, "MFGR#2221", Predicate::gte) && evaluatePredicate(reg_part__p_brand1, "MFGR#2228", Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_595111dc24f0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_595111dc24f0 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_595111dc24f0 = atomicAdd((int*)BUF_IDX_595111dc24f0, 1);
HT_595111dc24f0.insert(cuco::pair{KEY_595111dc24f0, buf_idx_595111dc24f0});
BUF_595111dc24f0[buf_idx_595111dc24f0 * 1 + 0] = tid;
}
__global__ void count_595111dce8d0(uint64_t* COUNT595111dbed20, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT595111dbed20, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_595111dce8d0(uint64_t* BUF_595111dbed20, uint64_t* BUF_IDX_595111dbed20, HASHTABLE_INSERT HT_595111dbed20, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_595111dbed20 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_595111dbed20 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_595111dbed20 = atomicAdd((int*)BUF_IDX_595111dbed20, 1);
HT_595111dbed20.insert(cuco::pair{KEY_595111dbed20, buf_idx_595111dbed20});
BUF_595111dbed20[buf_idx_595111dbed20 * 1 + 0] = tid;
}
__global__ void count_595111da63b0(uint64_t* COUNT595111dc0ed0, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT595111dc0ed0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_595111da63b0(uint64_t* BUF_595111dc0ed0, uint64_t* BUF_IDX_595111dc0ed0, HASHTABLE_INSERT HT_595111dc0ed0, DBI32Type* date__d_datekey, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_595111dc0ed0 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_595111dc0ed0 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_595111dc0ed0 = atomicAdd((int*)BUF_IDX_595111dc0ed0, 1);
HT_595111dc0ed0.insert(cuco::pair{KEY_595111dc0ed0, buf_idx_595111dc0ed0});
BUF_595111dc0ed0[buf_idx_595111dc0ed0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_595111da5de0(uint64_t* BUF_595111dbed20, uint64_t* BUF_595111dc0ed0, uint64_t* BUF_595111dc24f0, HASHTABLE_INSERT HT_595111d75620, HASHTABLE_PROBE HT_595111dbed20, HASHTABLE_PROBE HT_595111dc0ed0, HASHTABLE_PROBE HT_595111dc24f0, DBI32Type* date__d_year, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_595111dc24f0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_595111dc24f0 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_595111dc24f0.for_each(KEY_595111dc24f0, [&] __device__ (auto const SLOT_595111dc24f0) {

auto const [slot_first595111dc24f0, slot_second595111dc24f0] = SLOT_595111dc24f0;
if (!(true)) return;
uint64_t KEY_595111dbed20 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_595111dbed20 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_595111dbed20.for_each(KEY_595111dbed20, [&] __device__ (auto const SLOT_595111dbed20) {

auto const [slot_first595111dbed20, slot_second595111dbed20] = SLOT_595111dbed20;
if (!(true)) return;
uint64_t KEY_595111dc0ed0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_595111dc0ed0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_595111dc0ed0.for_each(KEY_595111dc0ed0, [&] __device__ (auto const SLOT_595111dc0ed0) {

auto const [slot_first595111dc0ed0, slot_second595111dc0ed0] = SLOT_595111dc0ed0;
if (!(true)) return;
uint64_t KEY_595111d75620 = 0;
auto reg_date__d_year = date__d_year[BUF_595111dc0ed0[slot_second595111dc0ed0 * 1 + 0]];

KEY_595111d75620 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_595111dc24f0[slot_second595111dc24f0 * 1 + 0]];
KEY_595111d75620 <<= 16;
KEY_595111d75620 |= reg_part__p_brand1_encoded;
//Create aggregation hash table
HT_595111d75620.insert(cuco::pair{KEY_595111d75620, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_595111da5de0(uint64_t* BUF_595111dbed20, uint64_t* BUF_595111dc0ed0, uint64_t* BUF_595111dc24f0, HASHTABLE_FIND HT_595111d75620, HASHTABLE_PROBE HT_595111dbed20, HASHTABLE_PROBE HT_595111dc0ed0, HASHTABLE_PROBE HT_595111dc24f0, DBI32Type* KEY_595111d75620date__d_year, DBI16Type* KEY_595111d75620part__p_brand1_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_595111dc24f0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_595111dc24f0 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_595111dc24f0.for_each(KEY_595111dc24f0, [&] __device__ (auto const SLOT_595111dc24f0) {
auto const [slot_first595111dc24f0, slot_second595111dc24f0] = SLOT_595111dc24f0;
if (!(true)) return;
uint64_t KEY_595111dbed20 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_595111dbed20 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_595111dbed20.for_each(KEY_595111dbed20, [&] __device__ (auto const SLOT_595111dbed20) {
auto const [slot_first595111dbed20, slot_second595111dbed20] = SLOT_595111dbed20;
if (!(true)) return;
uint64_t KEY_595111dc0ed0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_595111dc0ed0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_595111dc0ed0.for_each(KEY_595111dc0ed0, [&] __device__ (auto const SLOT_595111dc0ed0) {
auto const [slot_first595111dc0ed0, slot_second595111dc0ed0] = SLOT_595111dc0ed0;
if (!(true)) return;
uint64_t KEY_595111d75620 = 0;
auto reg_date__d_year = date__d_year[BUF_595111dc0ed0[slot_second595111dc0ed0 * 1 + 0]];

KEY_595111d75620 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_595111dc24f0[slot_second595111dc24f0 * 1 + 0]];
KEY_595111d75620 <<= 16;
KEY_595111d75620 |= reg_part__p_brand1_encoded;
//Aggregate in hashtable
auto buf_idx_595111d75620 = HT_595111d75620.find(KEY_595111d75620)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_595111d75620], reg_lineorder__lo_revenue);
KEY_595111d75620date__d_year[buf_idx_595111d75620] = reg_date__d_year;
KEY_595111d75620part__p_brand1_encoded[buf_idx_595111d75620] = reg_part__p_brand1_encoded;
});
});
});
}
__global__ void count_595111ddde40(size_t COUNT595111d75620, uint64_t* COUNT595111d89400) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT595111d75620) return;
//Materialize count
atomicAdd((int*)COUNT595111d89400, 1);
}
__global__ void main_595111ddde40(size_t COUNT595111d75620, DBDecimalType* MAT595111d89400aggr0__tmp_attr0, DBI32Type* MAT595111d89400date__d_year, DBI16Type* MAT595111d89400part__p_brand1_encoded, uint64_t* MAT_IDX595111d89400, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT595111d75620) return;
//Materialize buffers
auto mat_idx595111d89400 = atomicAdd((int*)MAT_IDX595111d89400, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT595111d89400aggr0__tmp_attr0[mat_idx595111d89400] = reg_aggr0__tmp_attr0;
auto reg_date__d_year = date__d_year[tid];
MAT595111d89400date__d_year[mat_idx595111d89400] = reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[tid];
MAT595111d89400part__p_brand1_encoded[mat_idx595111d89400] = reg_part__p_brand1_encoded;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT595111dc24f0;
hipMalloc(&d_COUNT595111dc24f0, sizeof(uint64_t));
hipMemset(d_COUNT595111dc24f0, 0, sizeof(uint64_t));
count_595111dcbfd0<<<std::ceil((float)part_size/128.), 128>>>(d_COUNT595111dc24f0, d_part__p_brand1, part_size);
uint64_t COUNT595111dc24f0;
hipMemcpy(&COUNT595111dc24f0, d_COUNT595111dc24f0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_595111dc24f0;
hipMalloc(&d_BUF_IDX_595111dc24f0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_595111dc24f0, 0, sizeof(uint64_t));
uint64_t* d_BUF_595111dc24f0;
hipMalloc(&d_BUF_595111dc24f0, sizeof(uint64_t) * COUNT595111dc24f0 * 1);
auto d_HT_595111dc24f0 = cuco::experimental::static_multimap{ (int)COUNT595111dc24f0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_595111dcbfd0<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_595111dc24f0, d_BUF_IDX_595111dc24f0, d_HT_595111dc24f0.ref(cuco::insert), d_part__p_brand1, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT595111dbed20;
hipMalloc(&d_COUNT595111dbed20, sizeof(uint64_t));
hipMemset(d_COUNT595111dbed20, 0, sizeof(uint64_t));
count_595111dce8d0<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT595111dbed20, d_supplier__s_region, supplier_size);
uint64_t COUNT595111dbed20;
hipMemcpy(&COUNT595111dbed20, d_COUNT595111dbed20, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_595111dbed20;
hipMalloc(&d_BUF_IDX_595111dbed20, sizeof(uint64_t));
hipMemset(d_BUF_IDX_595111dbed20, 0, sizeof(uint64_t));
uint64_t* d_BUF_595111dbed20;
hipMalloc(&d_BUF_595111dbed20, sizeof(uint64_t) * COUNT595111dbed20 * 1);
auto d_HT_595111dbed20 = cuco::experimental::static_multimap{ (int)COUNT595111dbed20*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_595111dce8d0<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_595111dbed20, d_BUF_IDX_595111dbed20, d_HT_595111dbed20.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT595111dc0ed0;
hipMalloc(&d_COUNT595111dc0ed0, sizeof(uint64_t));
hipMemset(d_COUNT595111dc0ed0, 0, sizeof(uint64_t));
count_595111da63b0<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT595111dc0ed0, date_size);
uint64_t COUNT595111dc0ed0;
hipMemcpy(&COUNT595111dc0ed0, d_COUNT595111dc0ed0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_595111dc0ed0;
hipMalloc(&d_BUF_IDX_595111dc0ed0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_595111dc0ed0, 0, sizeof(uint64_t));
uint64_t* d_BUF_595111dc0ed0;
hipMalloc(&d_BUF_595111dc0ed0, sizeof(uint64_t) * COUNT595111dc0ed0 * 1);
auto d_HT_595111dc0ed0 = cuco::experimental::static_multimap{ (int)COUNT595111dc0ed0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_595111da63b0<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_595111dc0ed0, d_BUF_IDX_595111dc0ed0, d_HT_595111dc0ed0.ref(cuco::insert), d_date__d_datekey, date_size);
//Create aggregation hash table
auto d_HT_595111d75620 = cuco::static_map{ (int)3846*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_595111da5de0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_595111dbed20, d_BUF_595111dc0ed0, d_BUF_595111dc24f0, d_HT_595111d75620.ref(cuco::insert), d_HT_595111dbed20.ref(cuco::for_each), d_HT_595111dc0ed0.ref(cuco::for_each), d_HT_595111dc24f0.ref(cuco::for_each), d_date__d_year, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size, d_part__p_brand1_encoded);
size_t COUNT595111d75620 = d_HT_595111d75620.size();
thrust::device_vector<int64_t> keys_595111d75620(COUNT595111d75620), vals_595111d75620(COUNT595111d75620);
d_HT_595111d75620.retrieve_all(keys_595111d75620.begin(), vals_595111d75620.begin());
d_HT_595111d75620.clear();
int64_t* raw_keys595111d75620 = thrust::raw_pointer_cast(keys_595111d75620.data());
insertKeys<<<std::ceil((float)COUNT595111d75620/128.), 128>>>(raw_keys595111d75620, d_HT_595111d75620.ref(cuco::insert), COUNT595111d75620);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT595111d75620);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT595111d75620);
DBI32Type* d_KEY_595111d75620date__d_year;
hipMalloc(&d_KEY_595111d75620date__d_year, sizeof(DBI32Type) * COUNT595111d75620);
hipMemset(d_KEY_595111d75620date__d_year, 0, sizeof(DBI32Type) * COUNT595111d75620);
DBI16Type* d_KEY_595111d75620part__p_brand1_encoded;
hipMalloc(&d_KEY_595111d75620part__p_brand1_encoded, sizeof(DBI16Type) * COUNT595111d75620);
hipMemset(d_KEY_595111d75620part__p_brand1_encoded, 0, sizeof(DBI16Type) * COUNT595111d75620);
main_595111da5de0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_595111dbed20, d_BUF_595111dc0ed0, d_BUF_595111dc24f0, d_HT_595111d75620.ref(cuco::find), d_HT_595111dbed20.ref(cuco::for_each), d_HT_595111dc0ed0.ref(cuco::for_each), d_HT_595111dc24f0.ref(cuco::for_each), d_KEY_595111d75620date__d_year, d_KEY_595111d75620part__p_brand1_encoded, d_aggr0__tmp_attr0, d_date__d_year, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_part__p_brand1_encoded);
//Materialize count
uint64_t* d_COUNT595111d89400;
hipMalloc(&d_COUNT595111d89400, sizeof(uint64_t));
hipMemset(d_COUNT595111d89400, 0, sizeof(uint64_t));
count_595111ddde40<<<std::ceil((float)COUNT595111d75620/128.), 128>>>(COUNT595111d75620, d_COUNT595111d89400);
uint64_t COUNT595111d89400;
hipMemcpy(&COUNT595111d89400, d_COUNT595111d89400, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX595111d89400;
hipMalloc(&d_MAT_IDX595111d89400, sizeof(uint64_t));
hipMemset(d_MAT_IDX595111d89400, 0, sizeof(uint64_t));
auto MAT595111d89400aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT595111d89400);
DBDecimalType* d_MAT595111d89400aggr0__tmp_attr0;
hipMalloc(&d_MAT595111d89400aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT595111d89400);
auto MAT595111d89400date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT595111d89400);
DBI32Type* d_MAT595111d89400date__d_year;
hipMalloc(&d_MAT595111d89400date__d_year, sizeof(DBI32Type) * COUNT595111d89400);
auto MAT595111d89400part__p_brand1_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT595111d89400);
DBI16Type* d_MAT595111d89400part__p_brand1_encoded;
hipMalloc(&d_MAT595111d89400part__p_brand1_encoded, sizeof(DBI16Type) * COUNT595111d89400);
main_595111ddde40<<<std::ceil((float)COUNT595111d75620/128.), 128>>>(COUNT595111d75620, d_MAT595111d89400aggr0__tmp_attr0, d_MAT595111d89400date__d_year, d_MAT595111d89400part__p_brand1_encoded, d_MAT_IDX595111d89400, d_aggr0__tmp_attr0, d_KEY_595111d75620date__d_year, d_KEY_595111d75620part__p_brand1_encoded);
hipMemcpy(MAT595111d89400aggr0__tmp_attr0, d_MAT595111d89400aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT595111d89400, hipMemcpyDeviceToHost);
hipMemcpy(MAT595111d89400date__d_year, d_MAT595111d89400date__d_year, sizeof(DBI32Type) * COUNT595111d89400, hipMemcpyDeviceToHost);
hipMemcpy(MAT595111d89400part__p_brand1_encoded, d_MAT595111d89400part__p_brand1_encoded, sizeof(DBI16Type) * COUNT595111d89400, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT595111d89400; i++) { std::cout << "" << MAT595111d89400aggr0__tmp_attr0[i];
std::cout << "," << MAT595111d89400date__d_year[i];
std::cout << "," << part__p_brand1_map[MAT595111d89400part__p_brand1_encoded[i]];
std::cout << std::endl; }
hipFree(d_BUF_595111dc24f0);
hipFree(d_BUF_IDX_595111dc24f0);
hipFree(d_COUNT595111dc24f0);
hipFree(d_BUF_595111dbed20);
hipFree(d_BUF_IDX_595111dbed20);
hipFree(d_COUNT595111dbed20);
hipFree(d_BUF_595111dc0ed0);
hipFree(d_BUF_IDX_595111dc0ed0);
hipFree(d_COUNT595111dc0ed0);
hipFree(d_KEY_595111d75620date__d_year);
hipFree(d_KEY_595111d75620part__p_brand1_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT595111d89400);
hipFree(d_MAT595111d89400aggr0__tmp_attr0);
hipFree(d_MAT595111d89400date__d_year);
hipFree(d_MAT595111d89400part__p_brand1_encoded);
hipFree(d_MAT_IDX595111d89400);
free(MAT595111d89400aggr0__tmp_attr0);
free(MAT595111d89400date__d_year);
free(MAT595111d89400part__p_brand1_encoded);
}