#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_INSERT>
__global__ void main_11(uint64_t* BUF_22, uint64_t* BUF_IDX_22, HASHTABLE_INSERT HT_22, int64_t* cycles_per_warp_main_11_join_build_22, int64_t* cycles_per_warp_main_11_selection_10, int64_t* cycles_per_warp_main_11_selection_12, int64_t* cycles_per_warp_main_11_selection_13, int64_t* cycles_per_warp_main_11_selection_14, int64_t* cycles_per_warp_main_11_selection_15, DBStringType* part__p_brand1, DBI32Type* part__p_partkey, size_t part_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_part__p_brand1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_brand1[ITEM] = part__p_brand1[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_part__p_brand1[ITEM], "MFGR#2221", Predicate::gte) && evaluatePredicate(reg_part__p_brand1[ITEM], "MFGR#2228", Predicate::lte);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_selection_10[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_selection_12[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_selection_13[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_selection_14[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_selection_15[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_22[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_22[ITEM] = 0;
KEY_22[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_22.insert(cuco::pair{KEY_22[ITEM], ITEM*TB + tid});
BUF_22[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_join_build_22[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT>
__global__ void main_17(uint64_t* BUF_23, uint64_t* BUF_IDX_23, HASHTABLE_INSERT HT_23, int64_t* cycles_per_warp_main_17_join_build_23, int64_t* cycles_per_warp_main_17_selection_16, int64_t* cycles_per_warp_main_17_selection_18, int64_t* cycles_per_warp_main_17_selection_19, int64_t* cycles_per_warp_main_17_selection_20, int64_t* cycles_per_warp_main_17_selection_21, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_supplier__s_region[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_region[ITEM] = supplier__s_region[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_supplier__s_region[ITEM], "ASIA", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_selection_16[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_selection_18[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_selection_19[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_selection_20[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_selection_21[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_23[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_23[ITEM] = 0;
KEY_23[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_23.insert(cuco::pair{KEY_23[ITEM], ITEM*TB + tid});
BUF_23[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_join_build_23[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT>
__global__ void main_8(uint64_t* BUF_24, uint64_t* BUF_IDX_24, HASHTABLE_INSERT HT_24, int64_t* cycles_per_warp_main_8_join_build_24, int64_t* cycles_per_warp_main_8_selection_7, int64_t* cycles_per_warp_main_8_selection_9, DBI32Type* date__d_datekey, size_t date_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_8_selection_7[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_8_selection_9[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_24[ITEMS_PER_THREAD];
DBI32Type reg_date__d_datekey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_datekey[ITEM] = date__d_datekey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_24[ITEM] = 0;
KEY_24[ITEM] |= reg_date__d_datekey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_24.insert(cuco::pair{KEY_24[ITEM], ITEM*TB + tid});
BUF_24[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_8_join_build_24[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_1(uint64_t* BUF_22, uint64_t* BUF_23, uint64_t* BUF_24, HASHTABLE_PROBE HT_22, HASHTABLE_PROBE HT_23, HASHTABLE_PROBE HT_24, HASHTABLE_FIND HT_25, DBI32Type* KEY_25date__d_year, DBI16Type* KEY_25part__p_brand1_encoded, int* SLOT_COUNT_25, DBDecimalType* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_1_aggregation_25, int64_t* cycles_per_warp_main_1_join_probe_22, int64_t* cycles_per_warp_main_1_join_probe_23, int64_t* cycles_per_warp_main_1_join_probe_24, int64_t* cycles_per_warp_main_1_selection_0, int64_t* cycles_per_warp_main_1_selection_2, int64_t* cycles_per_warp_main_1_selection_3, int64_t* cycles_per_warp_main_1_selection_4, int64_t* cycles_per_warp_main_1_selection_5, int64_t* cycles_per_warp_main_1_selection_6, DBI32Type* date__d_year, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* part__p_brand1_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_3[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_5[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_6[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_22[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_partkey[ITEM] = lineorder__lo_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_22[ITEM] = 0;
KEY_22[ITEM] |= reg_lineorder__lo_partkey[ITEM];
}
//Probe Hash table
int64_t slot_second22[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_22 = HT_22.find(KEY_22[ITEM]);
if (SLOT_22 == HT_22.end()) {selection_flags[ITEM] = 0; continue;}
slot_second22[ITEM] = SLOT_22->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_probe_22[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_23[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_suppkey[ITEM] = lineorder__lo_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_23[ITEM] = 0;
KEY_23[ITEM] |= reg_lineorder__lo_suppkey[ITEM];
}
//Probe Hash table
int64_t slot_second23[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_23 = HT_23.find(KEY_23[ITEM]);
if (SLOT_23 == HT_23.end()) {selection_flags[ITEM] = 0; continue;}
slot_second23[ITEM] = SLOT_23->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_probe_23[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_24[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_orderdate[ITEM] = lineorder__lo_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_24[ITEM] = 0;
KEY_24[ITEM] |= reg_lineorder__lo_orderdate[ITEM];
}
//Probe Hash table
int64_t slot_second24[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_24 = HT_24.find(KEY_24[ITEM]);
if (SLOT_24 == HT_24.end()) {selection_flags[ITEM] = 0; continue;}
slot_second24[ITEM] = SLOT_24->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_probe_24[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_25[ITEMS_PER_THREAD];
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_date__d_year[ITEM] = date__d_year[BUF_24[slot_second24[ITEM] * 1 + 0]];
}
DBI16Type reg_part__p_brand1_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_brand1_encoded[ITEM] = part__p_brand1_encoded[BUF_22[slot_second22[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_25[ITEM] = 0;
KEY_25[ITEM] |= reg_date__d_year[ITEM];
KEY_25[ITEM] <<= 16;
KEY_25[ITEM] |= reg_part__p_brand1_encoded[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineorder__lo_revenue[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_revenue[ITEM] = lineorder__lo_revenue[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_25 = get_aggregation_slot(KEY_25[ITEM], HT_25, SLOT_COUNT_25);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_25], reg_lineorder__lo_revenue[ITEM]);
KEY_25date__d_year[buf_idx_25] = reg_date__d_year[ITEM];
KEY_25part__p_brand1_encoded[buf_idx_25] = reg_part__p_brand1_encoded[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_aggregation_25[blockIdx.x] = cycles_per_warp;}
}
__global__ void main_27(size_t COUNT25, DBDecimalType* MAT26aggr0__tmp_attr0, DBI32Type* MAT26date__d_year, DBI16Type* MAT26part__p_brand1_encoded, uint64_t* MAT_IDX26, DBDecimalType* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_27_materialize_26, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT25); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT25); ++ITEM) {
reg_date__d_year[ITEM] = date__d_year[ITEM*TB + tid];
}
DBI16Type reg_part__p_brand1_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT25); ++ITEM) {
reg_part__p_brand1_encoded[ITEM] = part__p_brand1_encoded[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT25); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx26 = atomicAdd((int*)MAT_IDX26, 1);
MAT26aggr0__tmp_attr0[mat_idx26] = reg_aggr0__tmp_attr0[ITEM];
MAT26date__d_year[mat_idx26] = reg_date__d_year[ITEM];
MAT26part__p_brand1_encoded[mat_idx26] = reg_part__p_brand1_encoded[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_27_materialize_26[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_11_selection_10;
auto main_11_selection_10_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_selection_10, sizeof(int64_t) * main_11_selection_10_cpw_size);
hipMemset(d_cycles_per_warp_main_11_selection_10, -1, sizeof(int64_t) * main_11_selection_10_cpw_size);
int64_t* d_cycles_per_warp_main_11_selection_12;
auto main_11_selection_12_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_selection_12, sizeof(int64_t) * main_11_selection_12_cpw_size);
hipMemset(d_cycles_per_warp_main_11_selection_12, -1, sizeof(int64_t) * main_11_selection_12_cpw_size);
int64_t* d_cycles_per_warp_main_11_selection_13;
auto main_11_selection_13_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_selection_13, sizeof(int64_t) * main_11_selection_13_cpw_size);
hipMemset(d_cycles_per_warp_main_11_selection_13, -1, sizeof(int64_t) * main_11_selection_13_cpw_size);
int64_t* d_cycles_per_warp_main_11_selection_14;
auto main_11_selection_14_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_selection_14, sizeof(int64_t) * main_11_selection_14_cpw_size);
hipMemset(d_cycles_per_warp_main_11_selection_14, -1, sizeof(int64_t) * main_11_selection_14_cpw_size);
int64_t* d_cycles_per_warp_main_11_selection_15;
auto main_11_selection_15_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_selection_15, sizeof(int64_t) * main_11_selection_15_cpw_size);
hipMemset(d_cycles_per_warp_main_11_selection_15, -1, sizeof(int64_t) * main_11_selection_15_cpw_size);
int64_t* d_cycles_per_warp_main_11_join_build_22;
auto main_11_join_build_22_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_join_build_22, sizeof(int64_t) * main_11_join_build_22_cpw_size);
hipMemset(d_cycles_per_warp_main_11_join_build_22, -1, sizeof(int64_t) * main_11_join_build_22_cpw_size);
size_t COUNT22 = part_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_22;
hipMalloc(&d_BUF_IDX_22, sizeof(uint64_t));
hipMemset(d_BUF_IDX_22, 0, sizeof(uint64_t));
uint64_t* d_BUF_22;
hipMalloc(&d_BUF_22, sizeof(uint64_t) * COUNT22 * 1);
auto d_HT_22 = cuco::static_map{ (int)COUNT22*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_11<<<std::ceil((float)part_size/(float)TILE_SIZE), TB>>>(d_BUF_22, d_BUF_IDX_22, d_HT_22.ref(cuco::insert), d_cycles_per_warp_main_11_join_build_22, d_cycles_per_warp_main_11_selection_10, d_cycles_per_warp_main_11_selection_12, d_cycles_per_warp_main_11_selection_13, d_cycles_per_warp_main_11_selection_14, d_cycles_per_warp_main_11_selection_15, d_part__p_brand1, d_part__p_partkey, part_size);
int64_t* cycles_per_warp_main_11_selection_10 = (int64_t*)malloc(sizeof(int64_t) * main_11_selection_10_cpw_size);
hipMemcpy(cycles_per_warp_main_11_selection_10, d_cycles_per_warp_main_11_selection_10, sizeof(int64_t) * main_11_selection_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_selection_10 ";
for (auto i=0ull; i < main_11_selection_10_cpw_size; i++) std::cout << cycles_per_warp_main_11_selection_10[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_11_selection_12 = (int64_t*)malloc(sizeof(int64_t) * main_11_selection_12_cpw_size);
hipMemcpy(cycles_per_warp_main_11_selection_12, d_cycles_per_warp_main_11_selection_12, sizeof(int64_t) * main_11_selection_12_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_selection_12 ";
for (auto i=0ull; i < main_11_selection_12_cpw_size; i++) std::cout << cycles_per_warp_main_11_selection_12[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_11_selection_13 = (int64_t*)malloc(sizeof(int64_t) * main_11_selection_13_cpw_size);
hipMemcpy(cycles_per_warp_main_11_selection_13, d_cycles_per_warp_main_11_selection_13, sizeof(int64_t) * main_11_selection_13_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_selection_13 ";
for (auto i=0ull; i < main_11_selection_13_cpw_size; i++) std::cout << cycles_per_warp_main_11_selection_13[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_11_selection_14 = (int64_t*)malloc(sizeof(int64_t) * main_11_selection_14_cpw_size);
hipMemcpy(cycles_per_warp_main_11_selection_14, d_cycles_per_warp_main_11_selection_14, sizeof(int64_t) * main_11_selection_14_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_selection_14 ";
for (auto i=0ull; i < main_11_selection_14_cpw_size; i++) std::cout << cycles_per_warp_main_11_selection_14[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_11_selection_15 = (int64_t*)malloc(sizeof(int64_t) * main_11_selection_15_cpw_size);
hipMemcpy(cycles_per_warp_main_11_selection_15, d_cycles_per_warp_main_11_selection_15, sizeof(int64_t) * main_11_selection_15_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_selection_15 ";
for (auto i=0ull; i < main_11_selection_15_cpw_size; i++) std::cout << cycles_per_warp_main_11_selection_15[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_11_join_build_22 = (int64_t*)malloc(sizeof(int64_t) * main_11_join_build_22_cpw_size);
hipMemcpy(cycles_per_warp_main_11_join_build_22, d_cycles_per_warp_main_11_join_build_22, sizeof(int64_t) * main_11_join_build_22_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_join_build_22 ";
for (auto i=0ull; i < main_11_join_build_22_cpw_size; i++) std::cout << cycles_per_warp_main_11_join_build_22[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_17_selection_16;
auto main_17_selection_16_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_selection_16, sizeof(int64_t) * main_17_selection_16_cpw_size);
hipMemset(d_cycles_per_warp_main_17_selection_16, -1, sizeof(int64_t) * main_17_selection_16_cpw_size);
int64_t* d_cycles_per_warp_main_17_selection_18;
auto main_17_selection_18_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_selection_18, sizeof(int64_t) * main_17_selection_18_cpw_size);
hipMemset(d_cycles_per_warp_main_17_selection_18, -1, sizeof(int64_t) * main_17_selection_18_cpw_size);
int64_t* d_cycles_per_warp_main_17_selection_19;
auto main_17_selection_19_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_selection_19, sizeof(int64_t) * main_17_selection_19_cpw_size);
hipMemset(d_cycles_per_warp_main_17_selection_19, -1, sizeof(int64_t) * main_17_selection_19_cpw_size);
int64_t* d_cycles_per_warp_main_17_selection_20;
auto main_17_selection_20_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_selection_20, sizeof(int64_t) * main_17_selection_20_cpw_size);
hipMemset(d_cycles_per_warp_main_17_selection_20, -1, sizeof(int64_t) * main_17_selection_20_cpw_size);
int64_t* d_cycles_per_warp_main_17_selection_21;
auto main_17_selection_21_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_selection_21, sizeof(int64_t) * main_17_selection_21_cpw_size);
hipMemset(d_cycles_per_warp_main_17_selection_21, -1, sizeof(int64_t) * main_17_selection_21_cpw_size);
int64_t* d_cycles_per_warp_main_17_join_build_23;
auto main_17_join_build_23_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_join_build_23, sizeof(int64_t) * main_17_join_build_23_cpw_size);
hipMemset(d_cycles_per_warp_main_17_join_build_23, -1, sizeof(int64_t) * main_17_join_build_23_cpw_size);
size_t COUNT23 = supplier_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_23;
hipMalloc(&d_BUF_IDX_23, sizeof(uint64_t));
hipMemset(d_BUF_IDX_23, 0, sizeof(uint64_t));
uint64_t* d_BUF_23;
hipMalloc(&d_BUF_23, sizeof(uint64_t) * COUNT23 * 1);
auto d_HT_23 = cuco::static_map{ (int)COUNT23*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_17<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TB>>>(d_BUF_23, d_BUF_IDX_23, d_HT_23.ref(cuco::insert), d_cycles_per_warp_main_17_join_build_23, d_cycles_per_warp_main_17_selection_16, d_cycles_per_warp_main_17_selection_18, d_cycles_per_warp_main_17_selection_19, d_cycles_per_warp_main_17_selection_20, d_cycles_per_warp_main_17_selection_21, d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
int64_t* cycles_per_warp_main_17_selection_16 = (int64_t*)malloc(sizeof(int64_t) * main_17_selection_16_cpw_size);
hipMemcpy(cycles_per_warp_main_17_selection_16, d_cycles_per_warp_main_17_selection_16, sizeof(int64_t) * main_17_selection_16_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_selection_16 ";
for (auto i=0ull; i < main_17_selection_16_cpw_size; i++) std::cout << cycles_per_warp_main_17_selection_16[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_17_selection_18 = (int64_t*)malloc(sizeof(int64_t) * main_17_selection_18_cpw_size);
hipMemcpy(cycles_per_warp_main_17_selection_18, d_cycles_per_warp_main_17_selection_18, sizeof(int64_t) * main_17_selection_18_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_selection_18 ";
for (auto i=0ull; i < main_17_selection_18_cpw_size; i++) std::cout << cycles_per_warp_main_17_selection_18[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_17_selection_19 = (int64_t*)malloc(sizeof(int64_t) * main_17_selection_19_cpw_size);
hipMemcpy(cycles_per_warp_main_17_selection_19, d_cycles_per_warp_main_17_selection_19, sizeof(int64_t) * main_17_selection_19_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_selection_19 ";
for (auto i=0ull; i < main_17_selection_19_cpw_size; i++) std::cout << cycles_per_warp_main_17_selection_19[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_17_selection_20 = (int64_t*)malloc(sizeof(int64_t) * main_17_selection_20_cpw_size);
hipMemcpy(cycles_per_warp_main_17_selection_20, d_cycles_per_warp_main_17_selection_20, sizeof(int64_t) * main_17_selection_20_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_selection_20 ";
for (auto i=0ull; i < main_17_selection_20_cpw_size; i++) std::cout << cycles_per_warp_main_17_selection_20[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_17_selection_21 = (int64_t*)malloc(sizeof(int64_t) * main_17_selection_21_cpw_size);
hipMemcpy(cycles_per_warp_main_17_selection_21, d_cycles_per_warp_main_17_selection_21, sizeof(int64_t) * main_17_selection_21_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_selection_21 ";
for (auto i=0ull; i < main_17_selection_21_cpw_size; i++) std::cout << cycles_per_warp_main_17_selection_21[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_17_join_build_23 = (int64_t*)malloc(sizeof(int64_t) * main_17_join_build_23_cpw_size);
hipMemcpy(cycles_per_warp_main_17_join_build_23, d_cycles_per_warp_main_17_join_build_23, sizeof(int64_t) * main_17_join_build_23_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_join_build_23 ";
for (auto i=0ull; i < main_17_join_build_23_cpw_size; i++) std::cout << cycles_per_warp_main_17_join_build_23[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_8_selection_7;
auto main_8_selection_7_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_8_selection_7, sizeof(int64_t) * main_8_selection_7_cpw_size);
hipMemset(d_cycles_per_warp_main_8_selection_7, -1, sizeof(int64_t) * main_8_selection_7_cpw_size);
int64_t* d_cycles_per_warp_main_8_selection_9;
auto main_8_selection_9_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_8_selection_9, sizeof(int64_t) * main_8_selection_9_cpw_size);
hipMemset(d_cycles_per_warp_main_8_selection_9, -1, sizeof(int64_t) * main_8_selection_9_cpw_size);
int64_t* d_cycles_per_warp_main_8_join_build_24;
auto main_8_join_build_24_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_8_join_build_24, sizeof(int64_t) * main_8_join_build_24_cpw_size);
hipMemset(d_cycles_per_warp_main_8_join_build_24, -1, sizeof(int64_t) * main_8_join_build_24_cpw_size);
size_t COUNT24 = date_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_24;
hipMalloc(&d_BUF_IDX_24, sizeof(uint64_t));
hipMemset(d_BUF_IDX_24, 0, sizeof(uint64_t));
uint64_t* d_BUF_24;
hipMalloc(&d_BUF_24, sizeof(uint64_t) * COUNT24 * 1);
auto d_HT_24 = cuco::static_map{ (int)COUNT24*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_8<<<std::ceil((float)date_size/(float)TILE_SIZE), TB>>>(d_BUF_24, d_BUF_IDX_24, d_HT_24.ref(cuco::insert), d_cycles_per_warp_main_8_join_build_24, d_cycles_per_warp_main_8_selection_7, d_cycles_per_warp_main_8_selection_9, d_date__d_datekey, date_size);
int64_t* cycles_per_warp_main_8_selection_7 = (int64_t*)malloc(sizeof(int64_t) * main_8_selection_7_cpw_size);
hipMemcpy(cycles_per_warp_main_8_selection_7, d_cycles_per_warp_main_8_selection_7, sizeof(int64_t) * main_8_selection_7_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_8_selection_7 ";
for (auto i=0ull; i < main_8_selection_7_cpw_size; i++) std::cout << cycles_per_warp_main_8_selection_7[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_8_selection_9 = (int64_t*)malloc(sizeof(int64_t) * main_8_selection_9_cpw_size);
hipMemcpy(cycles_per_warp_main_8_selection_9, d_cycles_per_warp_main_8_selection_9, sizeof(int64_t) * main_8_selection_9_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_8_selection_9 ";
for (auto i=0ull; i < main_8_selection_9_cpw_size; i++) std::cout << cycles_per_warp_main_8_selection_9[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_8_join_build_24 = (int64_t*)malloc(sizeof(int64_t) * main_8_join_build_24_cpw_size);
hipMemcpy(cycles_per_warp_main_8_join_build_24, d_cycles_per_warp_main_8_join_build_24, sizeof(int64_t) * main_8_join_build_24_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_8_join_build_24 ";
for (auto i=0ull; i < main_8_join_build_24_cpw_size; i++) std::cout << cycles_per_warp_main_8_join_build_24[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_2;
auto main_1_selection_2_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_2, -1, sizeof(int64_t) * main_1_selection_2_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_3;
auto main_1_selection_3_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_3, -1, sizeof(int64_t) * main_1_selection_3_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_4;
auto main_1_selection_4_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_4, sizeof(int64_t) * main_1_selection_4_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_4, -1, sizeof(int64_t) * main_1_selection_4_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_5;
auto main_1_selection_5_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_5, sizeof(int64_t) * main_1_selection_5_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_5, -1, sizeof(int64_t) * main_1_selection_5_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_6;
auto main_1_selection_6_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_6, sizeof(int64_t) * main_1_selection_6_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_6, -1, sizeof(int64_t) * main_1_selection_6_cpw_size);
int64_t* d_cycles_per_warp_main_1_join_probe_22;
auto main_1_join_probe_22_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_probe_22, sizeof(int64_t) * main_1_join_probe_22_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_probe_22, -1, sizeof(int64_t) * main_1_join_probe_22_cpw_size);
int64_t* d_cycles_per_warp_main_1_join_probe_23;
auto main_1_join_probe_23_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_probe_23, sizeof(int64_t) * main_1_join_probe_23_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_probe_23, -1, sizeof(int64_t) * main_1_join_probe_23_cpw_size);
int64_t* d_cycles_per_warp_main_1_join_probe_24;
auto main_1_join_probe_24_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_probe_24, sizeof(int64_t) * main_1_join_probe_24_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_probe_24, -1, sizeof(int64_t) * main_1_join_probe_24_cpw_size);
int64_t* d_cycles_per_warp_main_1_aggregation_25;
auto main_1_aggregation_25_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_aggregation_25, sizeof(int64_t) * main_1_aggregation_25_cpw_size);
hipMemset(d_cycles_per_warp_main_1_aggregation_25, -1, sizeof(int64_t) * main_1_aggregation_25_cpw_size);
size_t COUNT25 = 3846;
auto d_HT_25 = cuco::static_map{ (int)3846*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_25;
hipMalloc(&d_SLOT_COUNT_25, sizeof(int));
hipMemset(d_SLOT_COUNT_25, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT25);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT25);
DBI32Type* d_KEY_25date__d_year;
hipMalloc(&d_KEY_25date__d_year, sizeof(DBI32Type) * COUNT25);
hipMemset(d_KEY_25date__d_year, 0, sizeof(DBI32Type) * COUNT25);
DBI16Type* d_KEY_25part__p_brand1_encoded;
hipMalloc(&d_KEY_25part__p_brand1_encoded, sizeof(DBI16Type) * COUNT25);
hipMemset(d_KEY_25part__p_brand1_encoded, 0, sizeof(DBI16Type) * COUNT25);
main_1<<<std::ceil((float)lineorder_size/(float)TILE_SIZE), TB>>>(d_BUF_22, d_BUF_23, d_BUF_24, d_HT_22.ref(cuco::find), d_HT_23.ref(cuco::find), d_HT_24.ref(cuco::find), d_HT_25.ref(cuco::insert_and_find), d_KEY_25date__d_year, d_KEY_25part__p_brand1_encoded, d_SLOT_COUNT_25, d_aggr0__tmp_attr0, d_cycles_per_warp_main_1_aggregation_25, d_cycles_per_warp_main_1_join_probe_22, d_cycles_per_warp_main_1_join_probe_23, d_cycles_per_warp_main_1_join_probe_24, d_cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_3, d_cycles_per_warp_main_1_selection_4, d_cycles_per_warp_main_1_selection_5, d_cycles_per_warp_main_1_selection_6, d_date__d_year, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_part__p_brand1_encoded);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_2 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_2 ";
for (auto i=0ull; i < main_1_selection_2_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_3 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_3, d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_3 ";
for (auto i=0ull; i < main_1_selection_3_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_3[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_4 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_4_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_4, d_cycles_per_warp_main_1_selection_4, sizeof(int64_t) * main_1_selection_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_4 ";
for (auto i=0ull; i < main_1_selection_4_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_5 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_5_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_5, d_cycles_per_warp_main_1_selection_5, sizeof(int64_t) * main_1_selection_5_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_5 ";
for (auto i=0ull; i < main_1_selection_5_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_5[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_6 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_6_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_6, d_cycles_per_warp_main_1_selection_6, sizeof(int64_t) * main_1_selection_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_6 ";
for (auto i=0ull; i < main_1_selection_6_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_6[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_probe_22 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_probe_22_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_probe_22, d_cycles_per_warp_main_1_join_probe_22, sizeof(int64_t) * main_1_join_probe_22_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_probe_22 ";
for (auto i=0ull; i < main_1_join_probe_22_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_probe_22[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_probe_23 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_probe_23_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_probe_23, d_cycles_per_warp_main_1_join_probe_23, sizeof(int64_t) * main_1_join_probe_23_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_probe_23 ";
for (auto i=0ull; i < main_1_join_probe_23_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_probe_23[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_probe_24 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_probe_24_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_probe_24, d_cycles_per_warp_main_1_join_probe_24, sizeof(int64_t) * main_1_join_probe_24_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_probe_24 ";
for (auto i=0ull; i < main_1_join_probe_24_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_probe_24[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_aggregation_25 = (int64_t*)malloc(sizeof(int64_t) * main_1_aggregation_25_cpw_size);
hipMemcpy(cycles_per_warp_main_1_aggregation_25, d_cycles_per_warp_main_1_aggregation_25, sizeof(int64_t) * main_1_aggregation_25_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_aggregation_25 ";
for (auto i=0ull; i < main_1_aggregation_25_cpw_size; i++) std::cout << cycles_per_warp_main_1_aggregation_25[i] << " ";
std::cout << std::endl;
COUNT25 = d_HT_25.size();
int64_t* d_cycles_per_warp_main_27_materialize_26;
auto main_27_materialize_26_cpw_size = std::ceil((float)COUNT25/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_27_materialize_26, sizeof(int64_t) * main_27_materialize_26_cpw_size);
hipMemset(d_cycles_per_warp_main_27_materialize_26, -1, sizeof(int64_t) * main_27_materialize_26_cpw_size);
size_t COUNT26 = COUNT25;
//Materialize buffers
uint64_t* d_MAT_IDX26;
hipMalloc(&d_MAT_IDX26, sizeof(uint64_t));
hipMemset(d_MAT_IDX26, 0, sizeof(uint64_t));
auto MAT26aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT26);
DBDecimalType* d_MAT26aggr0__tmp_attr0;
hipMalloc(&d_MAT26aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT26);
auto MAT26date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT26);
DBI32Type* d_MAT26date__d_year;
hipMalloc(&d_MAT26date__d_year, sizeof(DBI32Type) * COUNT26);
auto MAT26part__p_brand1_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT26);
DBI16Type* d_MAT26part__p_brand1_encoded;
hipMalloc(&d_MAT26part__p_brand1_encoded, sizeof(DBI16Type) * COUNT26);
main_27<<<std::ceil((float)COUNT25/(float)TILE_SIZE), TB>>>(COUNT25, d_MAT26aggr0__tmp_attr0, d_MAT26date__d_year, d_MAT26part__p_brand1_encoded, d_MAT_IDX26, d_aggr0__tmp_attr0, d_cycles_per_warp_main_27_materialize_26, d_KEY_25date__d_year, d_KEY_25part__p_brand1_encoded);
uint64_t MATCOUNT_26 = 0;
hipMemcpy(&MATCOUNT_26, d_MAT_IDX26, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT26aggr0__tmp_attr0, d_MAT26aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT26, hipMemcpyDeviceToHost);
hipMemcpy(MAT26date__d_year, d_MAT26date__d_year, sizeof(DBI32Type) * COUNT26, hipMemcpyDeviceToHost);
hipMemcpy(MAT26part__p_brand1_encoded, d_MAT26part__p_brand1_encoded, sizeof(DBI16Type) * COUNT26, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_27_materialize_26 = (int64_t*)malloc(sizeof(int64_t) * main_27_materialize_26_cpw_size);
hipMemcpy(cycles_per_warp_main_27_materialize_26, d_cycles_per_warp_main_27_materialize_26, sizeof(int64_t) * main_27_materialize_26_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_27_materialize_26 ";
for (auto i=0ull; i < main_27_materialize_26_cpw_size; i++) std::cout << cycles_per_warp_main_27_materialize_26[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_22);
hipFree(d_BUF_IDX_22);
hipFree(d_BUF_23);
hipFree(d_BUF_IDX_23);
hipFree(d_BUF_24);
hipFree(d_BUF_IDX_24);
hipFree(d_KEY_25date__d_year);
hipFree(d_KEY_25part__p_brand1_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_MAT26aggr0__tmp_attr0);
hipFree(d_MAT26date__d_year);
hipFree(d_MAT26part__p_brand1_encoded);
hipFree(d_MAT_IDX26);
free(MAT26aggr0__tmp_attr0);
free(MAT26date__d_year);
free(MAT26part__p_brand1_encoded);
}