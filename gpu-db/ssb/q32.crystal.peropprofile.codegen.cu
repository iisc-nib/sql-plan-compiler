#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_14(uint64_t* COUNT19, DBStringType* supplier__s_nation, size_t supplier_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_supplier__s_nation[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_nation[ITEM] = supplier__s_nation[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_supplier__s_nation[ITEM], "UNITED STATES", Predicate::eq);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT19, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_14(uint64_t* BUF_19, uint64_t* BUF_IDX_19, HASHTABLE_INSERT HT_19, int64_t* cycles_per_warp_main_14_join_build_19, int64_t* cycles_per_warp_main_14_selection_13, int64_t* cycles_per_warp_main_14_selection_15, int64_t* cycles_per_warp_main_14_selection_16, int64_t* cycles_per_warp_main_14_selection_17, int64_t* cycles_per_warp_main_14_selection_18, DBStringType* supplier__s_nation, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_supplier__s_nation[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_nation[ITEM] = supplier__s_nation[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_supplier__s_nation[ITEM], "UNITED STATES", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_14_selection_13[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_14_selection_15[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_14_selection_16[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_14_selection_17[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_14_selection_18[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_19[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_19[ITEM] = 0;
KEY_19[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_19 = atomicAdd((int*)BUF_IDX_19, 1);
HT_19.insert(cuco::pair{KEY_19[ITEM], buf_idx_19});
BUF_19[(buf_idx_19) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_14_join_build_19[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_1(uint64_t* COUNT20, DBStringType* customer__c_nation, size_t customer_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_customer__c_nation[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_nation[ITEM] = customer__c_nation[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_customer__c_nation[ITEM], "UNITED STATES", Predicate::eq);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT20, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_20, uint64_t* BUF_IDX_20, HASHTABLE_INSERT HT_20, DBI32Type* customer__c_custkey, DBStringType* customer__c_nation, size_t customer_size, int64_t* cycles_per_warp_main_1_join_build_20, int64_t* cycles_per_warp_main_1_selection_0, int64_t* cycles_per_warp_main_1_selection_2, int64_t* cycles_per_warp_main_1_selection_3, int64_t* cycles_per_warp_main_1_selection_4, int64_t* cycles_per_warp_main_1_selection_5) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_customer__c_nation[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_nation[ITEM] = customer__c_nation[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_customer__c_nation[ITEM], "UNITED STATES", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_3[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_5[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_20[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_20[ITEM] = 0;
KEY_20[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_20 = atomicAdd((int*)BUF_IDX_20, 1);
HT_20.insert(cuco::pair{KEY_20[ITEM], buf_idx_20});
BUF_20[(buf_idx_20) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_build_20[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_22(uint64_t* COUNT27, DBI32Type* date__d_year, size_t date_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_year[ITEM] = date__d_year[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_date__d_year[ITEM], 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year[ITEM], 1997, Predicate::lte);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT27, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_22(uint64_t* BUF_27, uint64_t* BUF_IDX_27, HASHTABLE_INSERT HT_27, int64_t* cycles_per_warp_main_22_join_build_27, int64_t* cycles_per_warp_main_22_selection_21, int64_t* cycles_per_warp_main_22_selection_23, int64_t* cycles_per_warp_main_22_selection_24, int64_t* cycles_per_warp_main_22_selection_25, int64_t* cycles_per_warp_main_22_selection_26, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_year[ITEM] = date__d_year[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_date__d_year[ITEM], 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year[ITEM], 1997, Predicate::lte);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_22_selection_21[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_22_selection_23[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_22_selection_24[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_22_selection_25[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_22_selection_26[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_27[ITEMS_PER_THREAD];
DBI32Type reg_date__d_datekey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_datekey[ITEM] = date__d_datekey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_27[ITEM] = 0;
KEY_27[ITEM] |= reg_date__d_datekey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_27 = atomicAdd((int*)BUF_IDX_27, 1);
HT_27.insert(cuco::pair{KEY_27[ITEM], buf_idx_27});
BUF_27[(buf_idx_27) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_22_join_build_27[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_7(uint64_t* BUF_19, uint64_t* BUF_20, uint64_t* BUF_27, HASHTABLE_PROBE HT_19, HASHTABLE_PROBE HT_20, HASHTABLE_PROBE HT_27, HASHTABLE_INSERT HT_28, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
uint64_t KEY_19[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_suppkey[ITEM] = lineorder__lo_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_19[ITEM] = 0;
KEY_19[ITEM] |= reg_lineorder__lo_suppkey[ITEM];
}
int64_t slot_second19[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_19 = HT_19.find(KEY_19[ITEM]);
if (SLOT_19 == HT_19.end()) {selection_flags[ITEM] = 0; continue;}
slot_second19[ITEM] = SLOT_19->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_20[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_custkey[ITEM] = lineorder__lo_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_20[ITEM] = 0;
KEY_20[ITEM] |= reg_lineorder__lo_custkey[ITEM];
}
int64_t slot_second20[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_20 = HT_20.find(KEY_20[ITEM]);
if (SLOT_20 == HT_20.end()) {selection_flags[ITEM] = 0; continue;}
slot_second20[ITEM] = SLOT_20->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_27[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_orderdate[ITEM] = lineorder__lo_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_27[ITEM] = 0;
KEY_27[ITEM] |= reg_lineorder__lo_orderdate[ITEM];
}
int64_t slot_second27[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_27 = HT_27.find(KEY_27[ITEM]);
if (SLOT_27 == HT_27.end()) {selection_flags[ITEM] = 0; continue;}
slot_second27[ITEM] = SLOT_27->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_28[ITEMS_PER_THREAD];
DBI16Type reg_customer__c_city_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_city_encoded[ITEM] = customer__c_city_encoded[BUF_20[slot_second20[ITEM] * 1 + 0]];
}
DBI16Type reg_supplier__s_city_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_supplier__s_city_encoded[ITEM] = supplier__s_city_encoded[BUF_19[slot_second19[ITEM] * 1 + 0]];
}
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_date__d_year[ITEM] = date__d_year[BUF_27[slot_second27[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_28[ITEM] = 0;
KEY_28[ITEM] |= reg_customer__c_city_encoded[ITEM];
KEY_28[ITEM] <<= 16;
KEY_28[ITEM] |= reg_supplier__s_city_encoded[ITEM];
KEY_28[ITEM] <<= 32;
KEY_28[ITEM] |= reg_date__d_year[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_28.insert(cuco::pair{KEY_28[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_7(uint64_t* BUF_19, uint64_t* BUF_20, uint64_t* BUF_27, HASHTABLE_PROBE HT_19, HASHTABLE_PROBE HT_20, HASHTABLE_PROBE HT_27, HASHTABLE_FIND HT_28, DBI16Type* KEY_28customer__c_city_encoded, DBI32Type* KEY_28date__d_year, DBI16Type* KEY_28supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, int64_t* cycles_per_warp_main_7_aggregation_28, int64_t* cycles_per_warp_main_7_join_probe_19, int64_t* cycles_per_warp_main_7_join_probe_20, int64_t* cycles_per_warp_main_7_join_probe_27, int64_t* cycles_per_warp_main_7_selection_10, int64_t* cycles_per_warp_main_7_selection_11, int64_t* cycles_per_warp_main_7_selection_12, int64_t* cycles_per_warp_main_7_selection_6, int64_t* cycles_per_warp_main_7_selection_8, int64_t* cycles_per_warp_main_7_selection_9, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_selection_6[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_selection_8[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_selection_9[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_selection_10[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_selection_11[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_selection_12[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_19[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_suppkey[ITEM] = lineorder__lo_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_19[ITEM] = 0;
KEY_19[ITEM] |= reg_lineorder__lo_suppkey[ITEM];
}
int64_t slot_second19[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_19 = HT_19.find(KEY_19[ITEM]);
if (SLOT_19 == HT_19.end()) {selection_flags[ITEM] = 0; continue;}
slot_second19[ITEM] = SLOT_19->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_probe_19[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_20[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_custkey[ITEM] = lineorder__lo_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_20[ITEM] = 0;
KEY_20[ITEM] |= reg_lineorder__lo_custkey[ITEM];
}
int64_t slot_second20[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_20 = HT_20.find(KEY_20[ITEM]);
if (SLOT_20 == HT_20.end()) {selection_flags[ITEM] = 0; continue;}
slot_second20[ITEM] = SLOT_20->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_probe_20[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_27[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_orderdate[ITEM] = lineorder__lo_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_27[ITEM] = 0;
KEY_27[ITEM] |= reg_lineorder__lo_orderdate[ITEM];
}
int64_t slot_second27[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_27 = HT_27.find(KEY_27[ITEM]);
if (SLOT_27 == HT_27.end()) {selection_flags[ITEM] = 0; continue;}
slot_second27[ITEM] = SLOT_27->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_probe_27[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_28[ITEMS_PER_THREAD];
DBI16Type reg_customer__c_city_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_city_encoded[ITEM] = customer__c_city_encoded[BUF_20[slot_second20[ITEM] * 1 + 0]];
}
DBI16Type reg_supplier__s_city_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_supplier__s_city_encoded[ITEM] = supplier__s_city_encoded[BUF_19[slot_second19[ITEM] * 1 + 0]];
}
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_date__d_year[ITEM] = date__d_year[BUF_27[slot_second27[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_28[ITEM] = 0;
KEY_28[ITEM] |= reg_customer__c_city_encoded[ITEM];
KEY_28[ITEM] <<= 16;
KEY_28[ITEM] |= reg_supplier__s_city_encoded[ITEM];
KEY_28[ITEM] <<= 32;
KEY_28[ITEM] |= reg_date__d_year[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineorder__lo_revenue[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_revenue[ITEM] = lineorder__lo_revenue[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_28 = HT_28.find(KEY_28[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr0[buf_idx_28], reg_lineorder__lo_revenue[ITEM]);
KEY_28customer__c_city_encoded[buf_idx_28] = reg_customer__c_city_encoded[ITEM];
KEY_28supplier__s_city_encoded[buf_idx_28] = reg_supplier__s_city_encoded[ITEM];
KEY_28date__d_year[buf_idx_28] = reg_date__d_year[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_aggregation_28[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_30(size_t COUNT28, uint64_t* COUNT29) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT28); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT29, 1);
}
}
__global__ void main_30(size_t COUNT28, DBDecimalType* MAT29aggr0__tmp_attr0, DBI16Type* MAT29customer__c_city_encoded, DBI32Type* MAT29date__d_year, DBI16Type* MAT29supplier__s_city_encoded, uint64_t* MAT_IDX29, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, int64_t* cycles_per_warp_main_30_materialize_29, DBI32Type* date__d_year, DBI16Type* supplier__s_city_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBI16Type reg_customer__c_city_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT28); ++ITEM) {
reg_customer__c_city_encoded[ITEM] = customer__c_city_encoded[ITEM*TB + tid];
}
DBI16Type reg_supplier__s_city_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT28); ++ITEM) {
reg_supplier__s_city_encoded[ITEM] = supplier__s_city_encoded[ITEM*TB + tid];
}
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT28); ++ITEM) {
reg_date__d_year[ITEM] = date__d_year[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT28); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT28); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx29 = atomicAdd((int*)MAT_IDX29, 1);
MAT29customer__c_city_encoded[mat_idx29] = reg_customer__c_city_encoded[ITEM];
MAT29supplier__s_city_encoded[mat_idx29] = reg_supplier__s_city_encoded[ITEM];
MAT29date__d_year[mat_idx29] = reg_date__d_year[ITEM];
MAT29aggr0__tmp_attr0[mat_idx29] = reg_aggr0__tmp_attr0[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_30_materialize_29[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_14_selection_13;
auto main_14_selection_13_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_14_selection_13, sizeof(int64_t) * main_14_selection_13_cpw_size);
hipMemset(d_cycles_per_warp_main_14_selection_13, -1, sizeof(int64_t) * main_14_selection_13_cpw_size);
int64_t* d_cycles_per_warp_main_14_selection_15;
auto main_14_selection_15_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_14_selection_15, sizeof(int64_t) * main_14_selection_15_cpw_size);
hipMemset(d_cycles_per_warp_main_14_selection_15, -1, sizeof(int64_t) * main_14_selection_15_cpw_size);
int64_t* d_cycles_per_warp_main_14_selection_16;
auto main_14_selection_16_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_14_selection_16, sizeof(int64_t) * main_14_selection_16_cpw_size);
hipMemset(d_cycles_per_warp_main_14_selection_16, -1, sizeof(int64_t) * main_14_selection_16_cpw_size);
int64_t* d_cycles_per_warp_main_14_selection_17;
auto main_14_selection_17_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_14_selection_17, sizeof(int64_t) * main_14_selection_17_cpw_size);
hipMemset(d_cycles_per_warp_main_14_selection_17, -1, sizeof(int64_t) * main_14_selection_17_cpw_size);
int64_t* d_cycles_per_warp_main_14_selection_18;
auto main_14_selection_18_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_14_selection_18, sizeof(int64_t) * main_14_selection_18_cpw_size);
hipMemset(d_cycles_per_warp_main_14_selection_18, -1, sizeof(int64_t) * main_14_selection_18_cpw_size);
//Materialize count
uint64_t* d_COUNT19;
hipMalloc(&d_COUNT19, sizeof(uint64_t));
hipMemset(d_COUNT19, 0, sizeof(uint64_t));
count_14<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT19, d_supplier__s_nation, supplier_size);
uint64_t COUNT19;
hipMemcpy(&COUNT19, d_COUNT19, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_14_join_build_19;
auto main_14_join_build_19_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_14_join_build_19, sizeof(int64_t) * main_14_join_build_19_cpw_size);
hipMemset(d_cycles_per_warp_main_14_join_build_19, -1, sizeof(int64_t) * main_14_join_build_19_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_19;
hipMalloc(&d_BUF_IDX_19, sizeof(uint64_t));
hipMemset(d_BUF_IDX_19, 0, sizeof(uint64_t));
uint64_t* d_BUF_19;
hipMalloc(&d_BUF_19, sizeof(uint64_t) * COUNT19 * 1);
auto d_HT_19 = cuco::static_map{ (int)COUNT19*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_14<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_19, d_BUF_IDX_19, d_HT_19.ref(cuco::insert), d_cycles_per_warp_main_14_join_build_19, d_cycles_per_warp_main_14_selection_13, d_cycles_per_warp_main_14_selection_15, d_cycles_per_warp_main_14_selection_16, d_cycles_per_warp_main_14_selection_17, d_cycles_per_warp_main_14_selection_18, d_supplier__s_nation, d_supplier__s_suppkey, supplier_size);
int64_t* cycles_per_warp_main_14_selection_13 = (int64_t*)malloc(sizeof(int64_t) * main_14_selection_13_cpw_size);
hipMemcpy(cycles_per_warp_main_14_selection_13, d_cycles_per_warp_main_14_selection_13, sizeof(int64_t) * main_14_selection_13_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_14_selection_13 ";
for (auto i=0ull; i < main_14_selection_13_cpw_size; i++) std::cout << cycles_per_warp_main_14_selection_13[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_14_selection_15 = (int64_t*)malloc(sizeof(int64_t) * main_14_selection_15_cpw_size);
hipMemcpy(cycles_per_warp_main_14_selection_15, d_cycles_per_warp_main_14_selection_15, sizeof(int64_t) * main_14_selection_15_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_14_selection_15 ";
for (auto i=0ull; i < main_14_selection_15_cpw_size; i++) std::cout << cycles_per_warp_main_14_selection_15[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_14_selection_16 = (int64_t*)malloc(sizeof(int64_t) * main_14_selection_16_cpw_size);
hipMemcpy(cycles_per_warp_main_14_selection_16, d_cycles_per_warp_main_14_selection_16, sizeof(int64_t) * main_14_selection_16_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_14_selection_16 ";
for (auto i=0ull; i < main_14_selection_16_cpw_size; i++) std::cout << cycles_per_warp_main_14_selection_16[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_14_selection_17 = (int64_t*)malloc(sizeof(int64_t) * main_14_selection_17_cpw_size);
hipMemcpy(cycles_per_warp_main_14_selection_17, d_cycles_per_warp_main_14_selection_17, sizeof(int64_t) * main_14_selection_17_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_14_selection_17 ";
for (auto i=0ull; i < main_14_selection_17_cpw_size; i++) std::cout << cycles_per_warp_main_14_selection_17[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_14_selection_18 = (int64_t*)malloc(sizeof(int64_t) * main_14_selection_18_cpw_size);
hipMemcpy(cycles_per_warp_main_14_selection_18, d_cycles_per_warp_main_14_selection_18, sizeof(int64_t) * main_14_selection_18_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_14_selection_18 ";
for (auto i=0ull; i < main_14_selection_18_cpw_size; i++) std::cout << cycles_per_warp_main_14_selection_18[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_14_join_build_19 = (int64_t*)malloc(sizeof(int64_t) * main_14_join_build_19_cpw_size);
hipMemcpy(cycles_per_warp_main_14_join_build_19, d_cycles_per_warp_main_14_join_build_19, sizeof(int64_t) * main_14_join_build_19_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_14_join_build_19 ";
for (auto i=0ull; i < main_14_join_build_19_cpw_size; i++) std::cout << cycles_per_warp_main_14_join_build_19[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_2;
auto main_1_selection_2_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_2, -1, sizeof(int64_t) * main_1_selection_2_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_3;
auto main_1_selection_3_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_3, -1, sizeof(int64_t) * main_1_selection_3_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_4;
auto main_1_selection_4_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_4, sizeof(int64_t) * main_1_selection_4_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_4, -1, sizeof(int64_t) * main_1_selection_4_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_5;
auto main_1_selection_5_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_5, sizeof(int64_t) * main_1_selection_5_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_5, -1, sizeof(int64_t) * main_1_selection_5_cpw_size);
//Materialize count
uint64_t* d_COUNT20;
hipMalloc(&d_COUNT20, sizeof(uint64_t));
hipMemset(d_COUNT20, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT20, d_customer__c_nation, customer_size);
uint64_t COUNT20;
hipMemcpy(&COUNT20, d_COUNT20, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_1_join_build_20;
auto main_1_join_build_20_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_build_20, sizeof(int64_t) * main_1_join_build_20_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_build_20, -1, sizeof(int64_t) * main_1_join_build_20_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_20;
hipMalloc(&d_BUF_IDX_20, sizeof(uint64_t));
hipMemset(d_BUF_IDX_20, 0, sizeof(uint64_t));
uint64_t* d_BUF_20;
hipMalloc(&d_BUF_20, sizeof(uint64_t) * COUNT20 * 1);
auto d_HT_20 = cuco::static_map{ (int)COUNT20*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_20, d_BUF_IDX_20, d_HT_20.ref(cuco::insert), d_customer__c_custkey, d_customer__c_nation, customer_size, d_cycles_per_warp_main_1_join_build_20, d_cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_3, d_cycles_per_warp_main_1_selection_4, d_cycles_per_warp_main_1_selection_5);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_2 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_2 ";
for (auto i=0ull; i < main_1_selection_2_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_3 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_3, d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_3 ";
for (auto i=0ull; i < main_1_selection_3_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_3[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_4 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_4_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_4, d_cycles_per_warp_main_1_selection_4, sizeof(int64_t) * main_1_selection_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_4 ";
for (auto i=0ull; i < main_1_selection_4_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_5 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_5_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_5, d_cycles_per_warp_main_1_selection_5, sizeof(int64_t) * main_1_selection_5_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_5 ";
for (auto i=0ull; i < main_1_selection_5_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_5[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_build_20 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_build_20_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_build_20, d_cycles_per_warp_main_1_join_build_20, sizeof(int64_t) * main_1_join_build_20_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_build_20 ";
for (auto i=0ull; i < main_1_join_build_20_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_build_20[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_22_selection_21;
auto main_22_selection_21_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_22_selection_21, sizeof(int64_t) * main_22_selection_21_cpw_size);
hipMemset(d_cycles_per_warp_main_22_selection_21, -1, sizeof(int64_t) * main_22_selection_21_cpw_size);
int64_t* d_cycles_per_warp_main_22_selection_23;
auto main_22_selection_23_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_22_selection_23, sizeof(int64_t) * main_22_selection_23_cpw_size);
hipMemset(d_cycles_per_warp_main_22_selection_23, -1, sizeof(int64_t) * main_22_selection_23_cpw_size);
int64_t* d_cycles_per_warp_main_22_selection_24;
auto main_22_selection_24_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_22_selection_24, sizeof(int64_t) * main_22_selection_24_cpw_size);
hipMemset(d_cycles_per_warp_main_22_selection_24, -1, sizeof(int64_t) * main_22_selection_24_cpw_size);
int64_t* d_cycles_per_warp_main_22_selection_25;
auto main_22_selection_25_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_22_selection_25, sizeof(int64_t) * main_22_selection_25_cpw_size);
hipMemset(d_cycles_per_warp_main_22_selection_25, -1, sizeof(int64_t) * main_22_selection_25_cpw_size);
int64_t* d_cycles_per_warp_main_22_selection_26;
auto main_22_selection_26_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_22_selection_26, sizeof(int64_t) * main_22_selection_26_cpw_size);
hipMemset(d_cycles_per_warp_main_22_selection_26, -1, sizeof(int64_t) * main_22_selection_26_cpw_size);
//Materialize count
uint64_t* d_COUNT27;
hipMalloc(&d_COUNT27, sizeof(uint64_t));
hipMemset(d_COUNT27, 0, sizeof(uint64_t));
count_22<<<std::ceil((float)date_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT27, d_date__d_year, date_size);
uint64_t COUNT27;
hipMemcpy(&COUNT27, d_COUNT27, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_22_join_build_27;
auto main_22_join_build_27_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_22_join_build_27, sizeof(int64_t) * main_22_join_build_27_cpw_size);
hipMemset(d_cycles_per_warp_main_22_join_build_27, -1, sizeof(int64_t) * main_22_join_build_27_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_27;
hipMalloc(&d_BUF_IDX_27, sizeof(uint64_t));
hipMemset(d_BUF_IDX_27, 0, sizeof(uint64_t));
uint64_t* d_BUF_27;
hipMalloc(&d_BUF_27, sizeof(uint64_t) * COUNT27 * 1);
auto d_HT_27 = cuco::static_map{ (int)COUNT27*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_22<<<std::ceil((float)date_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_27, d_BUF_IDX_27, d_HT_27.ref(cuco::insert), d_cycles_per_warp_main_22_join_build_27, d_cycles_per_warp_main_22_selection_21, d_cycles_per_warp_main_22_selection_23, d_cycles_per_warp_main_22_selection_24, d_cycles_per_warp_main_22_selection_25, d_cycles_per_warp_main_22_selection_26, d_date__d_datekey, d_date__d_year, date_size);
int64_t* cycles_per_warp_main_22_selection_21 = (int64_t*)malloc(sizeof(int64_t) * main_22_selection_21_cpw_size);
hipMemcpy(cycles_per_warp_main_22_selection_21, d_cycles_per_warp_main_22_selection_21, sizeof(int64_t) * main_22_selection_21_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_22_selection_21 ";
for (auto i=0ull; i < main_22_selection_21_cpw_size; i++) std::cout << cycles_per_warp_main_22_selection_21[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_22_selection_23 = (int64_t*)malloc(sizeof(int64_t) * main_22_selection_23_cpw_size);
hipMemcpy(cycles_per_warp_main_22_selection_23, d_cycles_per_warp_main_22_selection_23, sizeof(int64_t) * main_22_selection_23_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_22_selection_23 ";
for (auto i=0ull; i < main_22_selection_23_cpw_size; i++) std::cout << cycles_per_warp_main_22_selection_23[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_22_selection_24 = (int64_t*)malloc(sizeof(int64_t) * main_22_selection_24_cpw_size);
hipMemcpy(cycles_per_warp_main_22_selection_24, d_cycles_per_warp_main_22_selection_24, sizeof(int64_t) * main_22_selection_24_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_22_selection_24 ";
for (auto i=0ull; i < main_22_selection_24_cpw_size; i++) std::cout << cycles_per_warp_main_22_selection_24[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_22_selection_25 = (int64_t*)malloc(sizeof(int64_t) * main_22_selection_25_cpw_size);
hipMemcpy(cycles_per_warp_main_22_selection_25, d_cycles_per_warp_main_22_selection_25, sizeof(int64_t) * main_22_selection_25_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_22_selection_25 ";
for (auto i=0ull; i < main_22_selection_25_cpw_size; i++) std::cout << cycles_per_warp_main_22_selection_25[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_22_selection_26 = (int64_t*)malloc(sizeof(int64_t) * main_22_selection_26_cpw_size);
hipMemcpy(cycles_per_warp_main_22_selection_26, d_cycles_per_warp_main_22_selection_26, sizeof(int64_t) * main_22_selection_26_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_22_selection_26 ";
for (auto i=0ull; i < main_22_selection_26_cpw_size; i++) std::cout << cycles_per_warp_main_22_selection_26[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_22_join_build_27 = (int64_t*)malloc(sizeof(int64_t) * main_22_join_build_27_cpw_size);
hipMemcpy(cycles_per_warp_main_22_join_build_27, d_cycles_per_warp_main_22_join_build_27, sizeof(int64_t) * main_22_join_build_27_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_22_join_build_27 ";
for (auto i=0ull; i < main_22_join_build_27_cpw_size; i++) std::cout << cycles_per_warp_main_22_join_build_27[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_7_selection_6;
auto main_7_selection_6_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_selection_6, sizeof(int64_t) * main_7_selection_6_cpw_size);
hipMemset(d_cycles_per_warp_main_7_selection_6, -1, sizeof(int64_t) * main_7_selection_6_cpw_size);
int64_t* d_cycles_per_warp_main_7_selection_8;
auto main_7_selection_8_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_selection_8, sizeof(int64_t) * main_7_selection_8_cpw_size);
hipMemset(d_cycles_per_warp_main_7_selection_8, -1, sizeof(int64_t) * main_7_selection_8_cpw_size);
int64_t* d_cycles_per_warp_main_7_selection_9;
auto main_7_selection_9_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_selection_9, sizeof(int64_t) * main_7_selection_9_cpw_size);
hipMemset(d_cycles_per_warp_main_7_selection_9, -1, sizeof(int64_t) * main_7_selection_9_cpw_size);
int64_t* d_cycles_per_warp_main_7_selection_10;
auto main_7_selection_10_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_selection_10, sizeof(int64_t) * main_7_selection_10_cpw_size);
hipMemset(d_cycles_per_warp_main_7_selection_10, -1, sizeof(int64_t) * main_7_selection_10_cpw_size);
int64_t* d_cycles_per_warp_main_7_selection_11;
auto main_7_selection_11_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_selection_11, sizeof(int64_t) * main_7_selection_11_cpw_size);
hipMemset(d_cycles_per_warp_main_7_selection_11, -1, sizeof(int64_t) * main_7_selection_11_cpw_size);
int64_t* d_cycles_per_warp_main_7_selection_12;
auto main_7_selection_12_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_selection_12, sizeof(int64_t) * main_7_selection_12_cpw_size);
hipMemset(d_cycles_per_warp_main_7_selection_12, -1, sizeof(int64_t) * main_7_selection_12_cpw_size);
int64_t* d_cycles_per_warp_main_7_join_probe_19;
auto main_7_join_probe_19_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_probe_19, sizeof(int64_t) * main_7_join_probe_19_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_probe_19, -1, sizeof(int64_t) * main_7_join_probe_19_cpw_size);
int64_t* d_cycles_per_warp_main_7_join_probe_20;
auto main_7_join_probe_20_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_probe_20, sizeof(int64_t) * main_7_join_probe_20_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_probe_20, -1, sizeof(int64_t) * main_7_join_probe_20_cpw_size);
int64_t* d_cycles_per_warp_main_7_join_probe_27;
auto main_7_join_probe_27_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_probe_27, sizeof(int64_t) * main_7_join_probe_27_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_probe_27, -1, sizeof(int64_t) * main_7_join_probe_27_cpw_size);
//Create aggregation hash table
auto d_HT_28 = cuco::static_map{ (int)5679*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_7<<<std::ceil((float)lineorder_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_19, d_BUF_20, d_BUF_27, d_HT_19.ref(cuco::find), d_HT_20.ref(cuco::find), d_HT_27.ref(cuco::find), d_HT_28.ref(cuco::insert), d_customer__c_city_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
size_t COUNT28 = d_HT_28.size();
thrust::device_vector<int64_t> keys_28(COUNT28), vals_28(COUNT28);
d_HT_28.retrieve_all(keys_28.begin(), vals_28.begin());
d_HT_28.clear();
int64_t* raw_keys28 = thrust::raw_pointer_cast(keys_28.data());
insertKeys<<<std::ceil((float)COUNT28/128.), 128>>>(raw_keys28, d_HT_28.ref(cuco::insert), COUNT28);
int64_t* d_cycles_per_warp_main_7_aggregation_28;
auto main_7_aggregation_28_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_aggregation_28, sizeof(int64_t) * main_7_aggregation_28_cpw_size);
hipMemset(d_cycles_per_warp_main_7_aggregation_28, -1, sizeof(int64_t) * main_7_aggregation_28_cpw_size);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT28);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT28);
DBI16Type* d_KEY_28customer__c_city_encoded;
hipMalloc(&d_KEY_28customer__c_city_encoded, sizeof(DBI16Type) * COUNT28);
hipMemset(d_KEY_28customer__c_city_encoded, 0, sizeof(DBI16Type) * COUNT28);
DBI16Type* d_KEY_28supplier__s_city_encoded;
hipMalloc(&d_KEY_28supplier__s_city_encoded, sizeof(DBI16Type) * COUNT28);
hipMemset(d_KEY_28supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT28);
DBI32Type* d_KEY_28date__d_year;
hipMalloc(&d_KEY_28date__d_year, sizeof(DBI32Type) * COUNT28);
hipMemset(d_KEY_28date__d_year, 0, sizeof(DBI32Type) * COUNT28);
main_7<<<std::ceil((float)lineorder_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_19, d_BUF_20, d_BUF_27, d_HT_19.ref(cuco::find), d_HT_20.ref(cuco::find), d_HT_27.ref(cuco::find), d_HT_28.ref(cuco::find), d_KEY_28customer__c_city_encoded, d_KEY_28date__d_year, d_KEY_28supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_city_encoded, d_cycles_per_warp_main_7_aggregation_28, d_cycles_per_warp_main_7_join_probe_19, d_cycles_per_warp_main_7_join_probe_20, d_cycles_per_warp_main_7_join_probe_27, d_cycles_per_warp_main_7_selection_10, d_cycles_per_warp_main_7_selection_11, d_cycles_per_warp_main_7_selection_12, d_cycles_per_warp_main_7_selection_6, d_cycles_per_warp_main_7_selection_8, d_cycles_per_warp_main_7_selection_9, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
int64_t* cycles_per_warp_main_7_selection_6 = (int64_t*)malloc(sizeof(int64_t) * main_7_selection_6_cpw_size);
hipMemcpy(cycles_per_warp_main_7_selection_6, d_cycles_per_warp_main_7_selection_6, sizeof(int64_t) * main_7_selection_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_selection_6 ";
for (auto i=0ull; i < main_7_selection_6_cpw_size; i++) std::cout << cycles_per_warp_main_7_selection_6[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_selection_8 = (int64_t*)malloc(sizeof(int64_t) * main_7_selection_8_cpw_size);
hipMemcpy(cycles_per_warp_main_7_selection_8, d_cycles_per_warp_main_7_selection_8, sizeof(int64_t) * main_7_selection_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_selection_8 ";
for (auto i=0ull; i < main_7_selection_8_cpw_size; i++) std::cout << cycles_per_warp_main_7_selection_8[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_selection_9 = (int64_t*)malloc(sizeof(int64_t) * main_7_selection_9_cpw_size);
hipMemcpy(cycles_per_warp_main_7_selection_9, d_cycles_per_warp_main_7_selection_9, sizeof(int64_t) * main_7_selection_9_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_selection_9 ";
for (auto i=0ull; i < main_7_selection_9_cpw_size; i++) std::cout << cycles_per_warp_main_7_selection_9[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_selection_10 = (int64_t*)malloc(sizeof(int64_t) * main_7_selection_10_cpw_size);
hipMemcpy(cycles_per_warp_main_7_selection_10, d_cycles_per_warp_main_7_selection_10, sizeof(int64_t) * main_7_selection_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_selection_10 ";
for (auto i=0ull; i < main_7_selection_10_cpw_size; i++) std::cout << cycles_per_warp_main_7_selection_10[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_selection_11 = (int64_t*)malloc(sizeof(int64_t) * main_7_selection_11_cpw_size);
hipMemcpy(cycles_per_warp_main_7_selection_11, d_cycles_per_warp_main_7_selection_11, sizeof(int64_t) * main_7_selection_11_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_selection_11 ";
for (auto i=0ull; i < main_7_selection_11_cpw_size; i++) std::cout << cycles_per_warp_main_7_selection_11[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_selection_12 = (int64_t*)malloc(sizeof(int64_t) * main_7_selection_12_cpw_size);
hipMemcpy(cycles_per_warp_main_7_selection_12, d_cycles_per_warp_main_7_selection_12, sizeof(int64_t) * main_7_selection_12_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_selection_12 ";
for (auto i=0ull; i < main_7_selection_12_cpw_size; i++) std::cout << cycles_per_warp_main_7_selection_12[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_join_probe_19 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_probe_19_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_probe_19, d_cycles_per_warp_main_7_join_probe_19, sizeof(int64_t) * main_7_join_probe_19_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_probe_19 ";
for (auto i=0ull; i < main_7_join_probe_19_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_probe_19[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_join_probe_20 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_probe_20_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_probe_20, d_cycles_per_warp_main_7_join_probe_20, sizeof(int64_t) * main_7_join_probe_20_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_probe_20 ";
for (auto i=0ull; i < main_7_join_probe_20_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_probe_20[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_join_probe_27 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_probe_27_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_probe_27, d_cycles_per_warp_main_7_join_probe_27, sizeof(int64_t) * main_7_join_probe_27_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_probe_27 ";
for (auto i=0ull; i < main_7_join_probe_27_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_probe_27[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_aggregation_28 = (int64_t*)malloc(sizeof(int64_t) * main_7_aggregation_28_cpw_size);
hipMemcpy(cycles_per_warp_main_7_aggregation_28, d_cycles_per_warp_main_7_aggregation_28, sizeof(int64_t) * main_7_aggregation_28_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_aggregation_28 ";
for (auto i=0ull; i < main_7_aggregation_28_cpw_size; i++) std::cout << cycles_per_warp_main_7_aggregation_28[i] << " ";
std::cout << std::endl;
//Materialize count
uint64_t* d_COUNT29;
hipMalloc(&d_COUNT29, sizeof(uint64_t));
hipMemset(d_COUNT29, 0, sizeof(uint64_t));
count_30<<<std::ceil((float)COUNT28/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT28, d_COUNT29);
uint64_t COUNT29;
hipMemcpy(&COUNT29, d_COUNT29, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_30_materialize_29;
auto main_30_materialize_29_cpw_size = std::ceil((float)COUNT28/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_30_materialize_29, sizeof(int64_t) * main_30_materialize_29_cpw_size);
hipMemset(d_cycles_per_warp_main_30_materialize_29, -1, sizeof(int64_t) * main_30_materialize_29_cpw_size);
//Materialize buffers
uint64_t* d_MAT_IDX29;
hipMalloc(&d_MAT_IDX29, sizeof(uint64_t));
hipMemset(d_MAT_IDX29, 0, sizeof(uint64_t));
auto MAT29customer__c_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT29);
DBI16Type* d_MAT29customer__c_city_encoded;
hipMalloc(&d_MAT29customer__c_city_encoded, sizeof(DBI16Type) * COUNT29);
auto MAT29supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT29);
DBI16Type* d_MAT29supplier__s_city_encoded;
hipMalloc(&d_MAT29supplier__s_city_encoded, sizeof(DBI16Type) * COUNT29);
auto MAT29date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT29);
DBI32Type* d_MAT29date__d_year;
hipMalloc(&d_MAT29date__d_year, sizeof(DBI32Type) * COUNT29);
auto MAT29aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT29);
DBDecimalType* d_MAT29aggr0__tmp_attr0;
hipMalloc(&d_MAT29aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT29);
main_30<<<std::ceil((float)COUNT28/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT28, d_MAT29aggr0__tmp_attr0, d_MAT29customer__c_city_encoded, d_MAT29date__d_year, d_MAT29supplier__s_city_encoded, d_MAT_IDX29, d_aggr0__tmp_attr0, d_KEY_28customer__c_city_encoded, d_cycles_per_warp_main_30_materialize_29, d_KEY_28date__d_year, d_KEY_28supplier__s_city_encoded);
hipMemcpy(MAT29customer__c_city_encoded, d_MAT29customer__c_city_encoded, sizeof(DBI16Type) * COUNT29, hipMemcpyDeviceToHost);
hipMemcpy(MAT29supplier__s_city_encoded, d_MAT29supplier__s_city_encoded, sizeof(DBI16Type) * COUNT29, hipMemcpyDeviceToHost);
hipMemcpy(MAT29date__d_year, d_MAT29date__d_year, sizeof(DBI32Type) * COUNT29, hipMemcpyDeviceToHost);
hipMemcpy(MAT29aggr0__tmp_attr0, d_MAT29aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT29, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_30_materialize_29 = (int64_t*)malloc(sizeof(int64_t) * main_30_materialize_29_cpw_size);
hipMemcpy(cycles_per_warp_main_30_materialize_29, d_cycles_per_warp_main_30_materialize_29, sizeof(int64_t) * main_30_materialize_29_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_30_materialize_29 ";
for (auto i=0ull; i < main_30_materialize_29_cpw_size; i++) std::cout << cycles_per_warp_main_30_materialize_29[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
      size_t aux_mem = usedGpuMem() - used_mem;
      std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_19);
hipFree(d_BUF_IDX_19);
hipFree(d_COUNT19);
hipFree(d_BUF_20);
hipFree(d_BUF_IDX_20);
hipFree(d_COUNT20);
hipFree(d_BUF_27);
hipFree(d_BUF_IDX_27);
hipFree(d_COUNT27);
hipFree(d_KEY_28customer__c_city_encoded);
hipFree(d_KEY_28date__d_year);
hipFree(d_KEY_28supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT29);
hipFree(d_MAT29aggr0__tmp_attr0);
hipFree(d_MAT29customer__c_city_encoded);
hipFree(d_MAT29date__d_year);
hipFree(d_MAT29supplier__s_city_encoded);
hipFree(d_MAT_IDX29);
free(MAT29aggr0__tmp_attr0);
free(MAT29customer__c_city_encoded);
free(MAT29date__d_year);
free(MAT29supplier__s_city_encoded);
}