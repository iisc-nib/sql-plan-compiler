#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_56c149ca99b0(uint64_t* COUNT56c149cc8bd0, DBStringType* supplier__s_nation, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT56c149cc8bd0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_56c149ca99b0(uint64_t* BUF_56c149cc8bd0, uint64_t* BUF_IDX_56c149cc8bd0, HASHTABLE_INSERT HT_56c149cc8bd0, DBStringType* supplier__s_nation, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_56c149cc8bd0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_56c149cc8bd0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_56c149cc8bd0 = atomicAdd((int*)BUF_IDX_56c149cc8bd0, 1);
HT_56c149cc8bd0.insert(cuco::pair{KEY_56c149cc8bd0, buf_idx_56c149cc8bd0});
BUF_56c149cc8bd0[buf_idx_56c149cc8bd0 * 1 + 0] = tid;
}
__global__ void count_56c149cc8700(uint64_t* COUNT56c149cc4d50, DBStringType* customer__c_nation, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_nation = customer__c_nation[tid];
if (!(evaluatePredicate(reg_customer__c_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT56c149cc4d50, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_56c149cc8700(uint64_t* BUF_56c149cc4d50, uint64_t* BUF_IDX_56c149cc4d50, HASHTABLE_INSERT HT_56c149cc4d50, DBI32Type* customer__c_custkey, DBStringType* customer__c_nation, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_nation = customer__c_nation[tid];
if (!(evaluatePredicate(reg_customer__c_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_56c149cc4d50 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_56c149cc4d50 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_56c149cc4d50 = atomicAdd((int*)BUF_IDX_56c149cc4d50, 1);
HT_56c149cc4d50.insert(cuco::pair{KEY_56c149cc4d50, buf_idx_56c149cc4d50});
BUF_56c149cc4d50[buf_idx_56c149cc4d50 * 1 + 0] = tid;
}
__global__ void count_56c149cd7060(uint64_t* COUNT56c149cc1e40, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT56c149cc1e40, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_56c149cd7060(uint64_t* BUF_56c149cc1e40, uint64_t* BUF_IDX_56c149cc1e40, HASHTABLE_INSERT HT_56c149cc1e40, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_56c149cc1e40 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_56c149cc1e40 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_56c149cc1e40 = atomicAdd((int*)BUF_IDX_56c149cc1e40, 1);
HT_56c149cc1e40.insert(cuco::pair{KEY_56c149cc1e40, buf_idx_56c149cc1e40});
BUF_56c149cc1e40[buf_idx_56c149cc1e40 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_56c149ca9410(uint64_t* BUF_56c149cc1e40, uint64_t* BUF_56c149cc4d50, uint64_t* BUF_56c149cc8bd0, HASHTABLE_INSERT HT_56c149c787b0, HASHTABLE_PROBE HT_56c149cc1e40, HASHTABLE_PROBE HT_56c149cc4d50, HASHTABLE_PROBE HT_56c149cc8bd0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_56c149cc8bd0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_56c149cc8bd0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_56c149cc8bd0 = HT_56c149cc8bd0.find(KEY_56c149cc8bd0);
if (SLOT_56c149cc8bd0 == HT_56c149cc8bd0.end()) return;
if (!(true)) return;
uint64_t KEY_56c149cc4d50 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_56c149cc4d50 |= reg_lineorder__lo_custkey;
//Probe Hash table
auto SLOT_56c149cc4d50 = HT_56c149cc4d50.find(KEY_56c149cc4d50);
if (SLOT_56c149cc4d50 == HT_56c149cc4d50.end()) return;
if (!(true)) return;
uint64_t KEY_56c149cc1e40 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_56c149cc1e40 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_56c149cc1e40 = HT_56c149cc1e40.find(KEY_56c149cc1e40);
if (SLOT_56c149cc1e40 == HT_56c149cc1e40.end()) return;
if (!(true)) return;
uint64_t KEY_56c149c787b0 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_56c149cc4d50[SLOT_56c149cc4d50->second * 1 + 0]];

KEY_56c149c787b0 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_56c149cc8bd0[SLOT_56c149cc8bd0->second * 1 + 0]];
KEY_56c149c787b0 <<= 16;
KEY_56c149c787b0 |= reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[BUF_56c149cc1e40[SLOT_56c149cc1e40->second * 1 + 0]];
KEY_56c149c787b0 <<= 32;
KEY_56c149c787b0 |= reg_date__d_year;
//Create aggregation hash table
HT_56c149c787b0.insert(cuco::pair{KEY_56c149c787b0, 1});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_56c149ca9410(uint64_t* BUF_56c149cc1e40, uint64_t* BUF_56c149cc4d50, uint64_t* BUF_56c149cc8bd0, HASHTABLE_FIND HT_56c149c787b0, HASHTABLE_PROBE HT_56c149cc1e40, HASHTABLE_PROBE HT_56c149cc4d50, HASHTABLE_PROBE HT_56c149cc8bd0, DBI16Type* KEY_56c149c787b0customer__c_city_encoded, DBI32Type* KEY_56c149c787b0date__d_year, DBI16Type* KEY_56c149c787b0supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_56c149cc8bd0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_56c149cc8bd0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_56c149cc8bd0 = HT_56c149cc8bd0.find(KEY_56c149cc8bd0);
if (SLOT_56c149cc8bd0 == HT_56c149cc8bd0.end()) return;
if (!(true)) return;
uint64_t KEY_56c149cc4d50 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_56c149cc4d50 |= reg_lineorder__lo_custkey;
//Probe Hash table
auto SLOT_56c149cc4d50 = HT_56c149cc4d50.find(KEY_56c149cc4d50);
if (SLOT_56c149cc4d50 == HT_56c149cc4d50.end()) return;
if (!(true)) return;
uint64_t KEY_56c149cc1e40 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_56c149cc1e40 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_56c149cc1e40 = HT_56c149cc1e40.find(KEY_56c149cc1e40);
if (SLOT_56c149cc1e40 == HT_56c149cc1e40.end()) return;
if (!(true)) return;
uint64_t KEY_56c149c787b0 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_56c149cc4d50[SLOT_56c149cc4d50->second * 1 + 0]];

KEY_56c149c787b0 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_56c149cc8bd0[SLOT_56c149cc8bd0->second * 1 + 0]];
KEY_56c149c787b0 <<= 16;
KEY_56c149c787b0 |= reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[BUF_56c149cc1e40[SLOT_56c149cc1e40->second * 1 + 0]];
KEY_56c149c787b0 <<= 32;
KEY_56c149c787b0 |= reg_date__d_year;
//Aggregate in hashtable
auto buf_idx_56c149c787b0 = HT_56c149c787b0.find(KEY_56c149c787b0)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_56c149c787b0], reg_lineorder__lo_revenue);
KEY_56c149c787b0customer__c_city_encoded[buf_idx_56c149c787b0] = reg_customer__c_city_encoded;
KEY_56c149c787b0supplier__s_city_encoded[buf_idx_56c149c787b0] = reg_supplier__s_city_encoded;
KEY_56c149c787b0date__d_year[buf_idx_56c149c787b0] = reg_date__d_year;
}
__global__ void count_56c149ce31f0(uint64_t* COUNT56c149c541e0, size_t COUNT56c149c787b0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT56c149c787b0) return;
//Materialize count
atomicAdd((int*)COUNT56c149c541e0, 1);
}
__global__ void main_56c149ce31f0(size_t COUNT56c149c787b0, DBDecimalType* MAT56c149c541e0aggr0__tmp_attr0, DBI16Type* MAT56c149c541e0customer__c_city_encoded, DBI32Type* MAT56c149c541e0date__d_year, DBI16Type* MAT56c149c541e0supplier__s_city_encoded, uint64_t* MAT_IDX56c149c541e0, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT56c149c787b0) return;
//Materialize buffers
auto mat_idx56c149c541e0 = atomicAdd((int*)MAT_IDX56c149c541e0, 1);
auto reg_customer__c_city_encoded = customer__c_city_encoded[tid];
MAT56c149c541e0customer__c_city_encoded[mat_idx56c149c541e0] = reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[tid];
MAT56c149c541e0supplier__s_city_encoded[mat_idx56c149c541e0] = reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[tid];
MAT56c149c541e0date__d_year[mat_idx56c149c541e0] = reg_date__d_year;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT56c149c541e0aggr0__tmp_attr0[mat_idx56c149c541e0] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT56c149cc8bd0;
hipMalloc(&d_COUNT56c149cc8bd0, sizeof(uint64_t));
hipMemset(d_COUNT56c149cc8bd0, 0, sizeof(uint64_t));
count_56c149ca99b0<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT56c149cc8bd0, d_supplier__s_nation, supplier_size);
uint64_t COUNT56c149cc8bd0;
hipMemcpy(&COUNT56c149cc8bd0, d_COUNT56c149cc8bd0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_56c149cc8bd0;
hipMalloc(&d_BUF_IDX_56c149cc8bd0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_56c149cc8bd0, 0, sizeof(uint64_t));
uint64_t* d_BUF_56c149cc8bd0;
hipMalloc(&d_BUF_56c149cc8bd0, sizeof(uint64_t) * COUNT56c149cc8bd0 * 1);
auto d_HT_56c149cc8bd0 = cuco::static_map{ (int)COUNT56c149cc8bd0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_56c149ca99b0<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_56c149cc8bd0, d_BUF_IDX_56c149cc8bd0, d_HT_56c149cc8bd0.ref(cuco::insert), d_supplier__s_nation, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT56c149cc4d50;
hipMalloc(&d_COUNT56c149cc4d50, sizeof(uint64_t));
hipMemset(d_COUNT56c149cc4d50, 0, sizeof(uint64_t));
count_56c149cc8700<<<std::ceil((float)customer_size/128.), 128>>>(d_COUNT56c149cc4d50, d_customer__c_nation, customer_size);
uint64_t COUNT56c149cc4d50;
hipMemcpy(&COUNT56c149cc4d50, d_COUNT56c149cc4d50, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_56c149cc4d50;
hipMalloc(&d_BUF_IDX_56c149cc4d50, sizeof(uint64_t));
hipMemset(d_BUF_IDX_56c149cc4d50, 0, sizeof(uint64_t));
uint64_t* d_BUF_56c149cc4d50;
hipMalloc(&d_BUF_56c149cc4d50, sizeof(uint64_t) * COUNT56c149cc4d50 * 1);
auto d_HT_56c149cc4d50 = cuco::static_map{ (int)COUNT56c149cc4d50*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_56c149cc8700<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_56c149cc4d50, d_BUF_IDX_56c149cc4d50, d_HT_56c149cc4d50.ref(cuco::insert), d_customer__c_custkey, d_customer__c_nation, customer_size);
//Materialize count
uint64_t* d_COUNT56c149cc1e40;
hipMalloc(&d_COUNT56c149cc1e40, sizeof(uint64_t));
hipMemset(d_COUNT56c149cc1e40, 0, sizeof(uint64_t));
count_56c149cd7060<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT56c149cc1e40, d_date__d_year, date_size);
uint64_t COUNT56c149cc1e40;
hipMemcpy(&COUNT56c149cc1e40, d_COUNT56c149cc1e40, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_56c149cc1e40;
hipMalloc(&d_BUF_IDX_56c149cc1e40, sizeof(uint64_t));
hipMemset(d_BUF_IDX_56c149cc1e40, 0, sizeof(uint64_t));
uint64_t* d_BUF_56c149cc1e40;
hipMalloc(&d_BUF_56c149cc1e40, sizeof(uint64_t) * COUNT56c149cc1e40 * 1);
auto d_HT_56c149cc1e40 = cuco::static_map{ (int)COUNT56c149cc1e40*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_56c149cd7060<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_56c149cc1e40, d_BUF_IDX_56c149cc1e40, d_HT_56c149cc1e40.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Create aggregation hash table
auto d_HT_56c149c787b0 = cuco::static_map{ (int)5679*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_56c149ca9410<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_56c149cc1e40, d_BUF_56c149cc4d50, d_BUF_56c149cc8bd0, d_HT_56c149c787b0.ref(cuco::insert), d_HT_56c149cc1e40.ref(cuco::find), d_HT_56c149cc4d50.ref(cuco::find), d_HT_56c149cc8bd0.ref(cuco::find), d_customer__c_city_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
size_t COUNT56c149c787b0 = d_HT_56c149c787b0.size();
thrust::device_vector<int64_t> keys_56c149c787b0(COUNT56c149c787b0), vals_56c149c787b0(COUNT56c149c787b0);
d_HT_56c149c787b0.retrieve_all(keys_56c149c787b0.begin(), vals_56c149c787b0.begin());
d_HT_56c149c787b0.clear();
int64_t* raw_keys56c149c787b0 = thrust::raw_pointer_cast(keys_56c149c787b0.data());
insertKeys<<<std::ceil((float)COUNT56c149c787b0/128.), 128>>>(raw_keys56c149c787b0, d_HT_56c149c787b0.ref(cuco::insert), COUNT56c149c787b0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT56c149c787b0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT56c149c787b0);
DBI16Type* d_KEY_56c149c787b0customer__c_city_encoded;
hipMalloc(&d_KEY_56c149c787b0customer__c_city_encoded, sizeof(DBI16Type) * COUNT56c149c787b0);
hipMemset(d_KEY_56c149c787b0customer__c_city_encoded, 0, sizeof(DBI16Type) * COUNT56c149c787b0);
DBI16Type* d_KEY_56c149c787b0supplier__s_city_encoded;
hipMalloc(&d_KEY_56c149c787b0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT56c149c787b0);
hipMemset(d_KEY_56c149c787b0supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT56c149c787b0);
DBI32Type* d_KEY_56c149c787b0date__d_year;
hipMalloc(&d_KEY_56c149c787b0date__d_year, sizeof(DBI32Type) * COUNT56c149c787b0);
hipMemset(d_KEY_56c149c787b0date__d_year, 0, sizeof(DBI32Type) * COUNT56c149c787b0);
main_56c149ca9410<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_56c149cc1e40, d_BUF_56c149cc4d50, d_BUF_56c149cc8bd0, d_HT_56c149c787b0.ref(cuco::find), d_HT_56c149cc1e40.ref(cuco::find), d_HT_56c149cc4d50.ref(cuco::find), d_HT_56c149cc8bd0.ref(cuco::find), d_KEY_56c149c787b0customer__c_city_encoded, d_KEY_56c149c787b0date__d_year, d_KEY_56c149c787b0supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_city_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
//Materialize count
uint64_t* d_COUNT56c149c541e0;
hipMalloc(&d_COUNT56c149c541e0, sizeof(uint64_t));
hipMemset(d_COUNT56c149c541e0, 0, sizeof(uint64_t));
count_56c149ce31f0<<<std::ceil((float)COUNT56c149c787b0/128.), 128>>>(d_COUNT56c149c541e0, COUNT56c149c787b0);
uint64_t COUNT56c149c541e0;
hipMemcpy(&COUNT56c149c541e0, d_COUNT56c149c541e0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX56c149c541e0;
hipMalloc(&d_MAT_IDX56c149c541e0, sizeof(uint64_t));
hipMemset(d_MAT_IDX56c149c541e0, 0, sizeof(uint64_t));
auto MAT56c149c541e0customer__c_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT56c149c541e0);
DBI16Type* d_MAT56c149c541e0customer__c_city_encoded;
hipMalloc(&d_MAT56c149c541e0customer__c_city_encoded, sizeof(DBI16Type) * COUNT56c149c541e0);
auto MAT56c149c541e0supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT56c149c541e0);
DBI16Type* d_MAT56c149c541e0supplier__s_city_encoded;
hipMalloc(&d_MAT56c149c541e0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT56c149c541e0);
auto MAT56c149c541e0date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT56c149c541e0);
DBI32Type* d_MAT56c149c541e0date__d_year;
hipMalloc(&d_MAT56c149c541e0date__d_year, sizeof(DBI32Type) * COUNT56c149c541e0);
auto MAT56c149c541e0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT56c149c541e0);
DBDecimalType* d_MAT56c149c541e0aggr0__tmp_attr0;
hipMalloc(&d_MAT56c149c541e0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT56c149c541e0);
main_56c149ce31f0<<<std::ceil((float)COUNT56c149c787b0/128.), 128>>>(COUNT56c149c787b0, d_MAT56c149c541e0aggr0__tmp_attr0, d_MAT56c149c541e0customer__c_city_encoded, d_MAT56c149c541e0date__d_year, d_MAT56c149c541e0supplier__s_city_encoded, d_MAT_IDX56c149c541e0, d_aggr0__tmp_attr0, d_KEY_56c149c787b0customer__c_city_encoded, d_KEY_56c149c787b0date__d_year, d_KEY_56c149c787b0supplier__s_city_encoded);
hipMemcpy(MAT56c149c541e0customer__c_city_encoded, d_MAT56c149c541e0customer__c_city_encoded, sizeof(DBI16Type) * COUNT56c149c541e0, hipMemcpyDeviceToHost);
hipMemcpy(MAT56c149c541e0supplier__s_city_encoded, d_MAT56c149c541e0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT56c149c541e0, hipMemcpyDeviceToHost);
hipMemcpy(MAT56c149c541e0date__d_year, d_MAT56c149c541e0date__d_year, sizeof(DBI32Type) * COUNT56c149c541e0, hipMemcpyDeviceToHost);
hipMemcpy(MAT56c149c541e0aggr0__tmp_attr0, d_MAT56c149c541e0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT56c149c541e0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT56c149c541e0; i++) { std::cout << "" << customer__c_city_map[MAT56c149c541e0customer__c_city_encoded[i]];
std::cout << "," << supplier__s_city_map[MAT56c149c541e0supplier__s_city_encoded[i]];
std::cout << "," << MAT56c149c541e0date__d_year[i];
std::cout << "," << MAT56c149c541e0aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_56c149cc8bd0);
hipFree(d_BUF_IDX_56c149cc8bd0);
hipFree(d_COUNT56c149cc8bd0);
hipFree(d_BUF_56c149cc4d50);
hipFree(d_BUF_IDX_56c149cc4d50);
hipFree(d_COUNT56c149cc4d50);
hipFree(d_BUF_56c149cc1e40);
hipFree(d_BUF_IDX_56c149cc1e40);
hipFree(d_COUNT56c149cc1e40);
hipFree(d_KEY_56c149c787b0customer__c_city_encoded);
hipFree(d_KEY_56c149c787b0date__d_year);
hipFree(d_KEY_56c149c787b0supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT56c149c541e0);
hipFree(d_MAT56c149c541e0aggr0__tmp_attr0);
hipFree(d_MAT56c149c541e0customer__c_city_encoded);
hipFree(d_MAT56c149c541e0date__d_year);
hipFree(d_MAT56c149c541e0supplier__s_city_encoded);
hipFree(d_MAT_IDX56c149c541e0);
free(MAT56c149c541e0aggr0__tmp_attr0);
free(MAT56c149c541e0customer__c_city_encoded);
free(MAT56c149c541e0date__d_year);
free(MAT56c149c541e0supplier__s_city_encoded);
}