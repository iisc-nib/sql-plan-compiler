#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
template<typename HASHTABLE_INSERT>
__global__ void count_1(HASHTABLE_INSERT HT_0, size_t lineorder_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x * ITEMS_PER_THREAD;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_0.insert(cuco::pair{KEY_0[ITEM], 1});
}
}
template<typename HASHTABLE_FIND>
__global__ void main_1(HASHTABLE_FIND HT_0, DBI64Type* aggr0__tmp_attr0, size_t lineorder_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x * ITEMS_PER_THREAD;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
}
//Aggregate in hashtable
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_0 = HT_0.find(KEY_0[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr0[buf_idx_0], 1);
}
}
__global__ void count_3(size_t COUNT0, uint64_t* COUNT2) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x * ITEMS_PER_THREAD;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT2, 1);
}
}
__global__ void main_3(size_t COUNT0, DBI64Type* MAT2aggr0__tmp_attr0, uint64_t* MAT_IDX2, DBI64Type* aggr0__tmp_attr0) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x * ITEMS_PER_THREAD;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize buffers
DBI64Type reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx2 = atomicAdd((int*)MAT_IDX2, 1);
MAT2aggr0__tmp_attr0[mat_idx2] = reg_aggr0__tmp_attr0[ITEM];
}
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Create aggregation hash table
auto d_HT_0 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_1<<<std::ceil((float)lineorder_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_HT_0.ref(cuco::insert), lineorder_size);
size_t COUNT0 = d_HT_0.size();
thrust::device_vector<int64_t> keys_0(COUNT0), vals_0(COUNT0);
d_HT_0.retrieve_all(keys_0.begin(), vals_0.begin());
d_HT_0.clear();
int64_t* raw_keys0 = thrust::raw_pointer_cast(keys_0.data());
insertKeys<<<std::ceil((float)COUNT0/128.), 128>>>(raw_keys0, d_HT_0.ref(cuco::insert), COUNT0);
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT0);
main_1<<<std::ceil((float)lineorder_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_HT_0.ref(cuco::find), d_aggr0__tmp_attr0, lineorder_size);
//Materialize count
uint64_t* d_COUNT2;
hipMalloc(&d_COUNT2, sizeof(uint64_t));
hipMemset(d_COUNT2, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)COUNT0/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT0, d_COUNT2);
uint64_t COUNT2;
hipMemcpy(&COUNT2, d_COUNT2, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX2;
hipMalloc(&d_MAT_IDX2, sizeof(uint64_t));
hipMemset(d_MAT_IDX2, 0, sizeof(uint64_t));
auto MAT2aggr0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT2);
DBI64Type* d_MAT2aggr0__tmp_attr0;
hipMalloc(&d_MAT2aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT2);
main_3<<<std::ceil((float)COUNT0/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT0, d_MAT2aggr0__tmp_attr0, d_MAT_IDX2, d_aggr0__tmp_attr0);
hipMemcpy(MAT2aggr0__tmp_attr0, d_MAT2aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT2, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT2; i++) { std::cout << "" << MAT2aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT2);
hipFree(d_MAT2aggr0__tmp_attr0);
hipFree(d_MAT_IDX2);
free(MAT2aggr0__tmp_attr0);
}