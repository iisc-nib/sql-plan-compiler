#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5e7cd62c1540(uint64_t* COUNT5e7cd62b1dd0, DBStringType* part__p_brand1, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_brand1 = part__p_brand1[tid];
if (!(evaluatePredicate(reg_part__p_brand1, "MFGR#2239", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5e7cd62b1dd0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5e7cd62c1540(uint64_t* BUF_5e7cd62b1dd0, uint64_t* BUF_IDX_5e7cd62b1dd0, HASHTABLE_INSERT HT_5e7cd62b1dd0, DBStringType* part__p_brand1, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_brand1 = part__p_brand1[tid];
if (!(evaluatePredicate(reg_part__p_brand1, "MFGR#2239", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e7cd62b1dd0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_5e7cd62b1dd0 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_5e7cd62b1dd0 = atomicAdd((int*)BUF_IDX_5e7cd62b1dd0, 1);
HT_5e7cd62b1dd0.insert(cuco::pair{KEY_5e7cd62b1dd0, buf_idx_5e7cd62b1dd0});
BUF_5e7cd62b1dd0[buf_idx_5e7cd62b1dd0 * 1 + 0] = tid;
}
__global__ void count_5e7cd62c39b0(uint64_t* COUNT5e7cd62b7280, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "EUROPE", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5e7cd62b7280, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5e7cd62c39b0(uint64_t* BUF_5e7cd62b7280, uint64_t* BUF_IDX_5e7cd62b7280, HASHTABLE_INSERT HT_5e7cd62b7280, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "EUROPE", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e7cd62b7280 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5e7cd62b7280 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5e7cd62b7280 = atomicAdd((int*)BUF_IDX_5e7cd62b7280, 1);
HT_5e7cd62b7280.insert(cuco::pair{KEY_5e7cd62b7280, buf_idx_5e7cd62b7280});
BUF_5e7cd62b7280[buf_idx_5e7cd62b7280 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5e7cd629a430(uint64_t* BUF_5e7cd62b1dd0, uint64_t* BUF_5e7cd62b7280, uint64_t* COUNT5e7cd62b1210, HASHTABLE_PROBE HT_5e7cd62b1dd0, HASHTABLE_PROBE HT_5e7cd62b7280, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e7cd62b1dd0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5e7cd62b1dd0 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5e7cd62b1dd0.for_each(KEY_5e7cd62b1dd0, [&] __device__ (auto const SLOT_5e7cd62b1dd0) {

auto const [slot_first5e7cd62b1dd0, slot_second5e7cd62b1dd0] = SLOT_5e7cd62b1dd0;
if (!(true)) return;
uint64_t KEY_5e7cd62b7280 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5e7cd62b7280 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5e7cd62b7280.for_each(KEY_5e7cd62b7280, [&] __device__ (auto const SLOT_5e7cd62b7280) {

auto const [slot_first5e7cd62b7280, slot_second5e7cd62b7280] = SLOT_5e7cd62b7280;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5e7cd62b1210, 1);
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_5e7cd629a430(uint64_t* BUF_5e7cd62b1210, uint64_t* BUF_5e7cd62b1dd0, uint64_t* BUF_5e7cd62b7280, uint64_t* BUF_IDX_5e7cd62b1210, HASHTABLE_INSERT HT_5e7cd62b1210, HASHTABLE_PROBE HT_5e7cd62b1dd0, HASHTABLE_PROBE HT_5e7cd62b7280, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e7cd62b1dd0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5e7cd62b1dd0 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5e7cd62b1dd0.for_each(KEY_5e7cd62b1dd0, [&] __device__ (auto const SLOT_5e7cd62b1dd0) {
auto const [slot_first5e7cd62b1dd0, slot_second5e7cd62b1dd0] = SLOT_5e7cd62b1dd0;
if (!(true)) return;
uint64_t KEY_5e7cd62b7280 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5e7cd62b7280 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5e7cd62b7280.for_each(KEY_5e7cd62b7280, [&] __device__ (auto const SLOT_5e7cd62b7280) {
auto const [slot_first5e7cd62b7280, slot_second5e7cd62b7280] = SLOT_5e7cd62b7280;
if (!(true)) return;
uint64_t KEY_5e7cd62b1210 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5e7cd62b1210 |= reg_lineorder__lo_orderdate;
// Insert hash table kernel;
auto buf_idx_5e7cd62b1210 = atomicAdd((int*)BUF_IDX_5e7cd62b1210, 1);
HT_5e7cd62b1210.insert(cuco::pair{KEY_5e7cd62b1210, buf_idx_5e7cd62b1210});
BUF_5e7cd62b1210[buf_idx_5e7cd62b1210 * 3 + 0] = tid;
BUF_5e7cd62b1210[buf_idx_5e7cd62b1210 * 3 + 1] = BUF_5e7cd62b1dd0[slot_second5e7cd62b1dd0 * 1 + 0];
BUF_5e7cd62b1210[buf_idx_5e7cd62b1210 * 3 + 2] = BUF_5e7cd62b7280[slot_second5e7cd62b7280 * 1 + 0];
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5e7cd629aa00(uint64_t* BUF_5e7cd62b1210, HASHTABLE_INSERT HT_5e7cd626a970, HASHTABLE_PROBE HT_5e7cd62b1210, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e7cd62b1210 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5e7cd62b1210 |= reg_date__d_datekey;
//Probe Hash table
HT_5e7cd62b1210.for_each(KEY_5e7cd62b1210, [&] __device__ (auto const SLOT_5e7cd62b1210) {

auto const [slot_first5e7cd62b1210, slot_second5e7cd62b1210] = SLOT_5e7cd62b1210;
if (!(true)) return;
uint64_t KEY_5e7cd626a970 = 0;
auto reg_date__d_year = date__d_year[tid];

KEY_5e7cd626a970 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_5e7cd62b1210[slot_second5e7cd62b1210 * 3 + 1]];
KEY_5e7cd626a970 <<= 16;
KEY_5e7cd626a970 |= reg_part__p_brand1_encoded;
//Create aggregation hash table
HT_5e7cd626a970.insert(cuco::pair{KEY_5e7cd626a970, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5e7cd629aa00(uint64_t* BUF_5e7cd62b1210, HASHTABLE_FIND HT_5e7cd626a970, HASHTABLE_PROBE HT_5e7cd62b1210, DBI32Type* KEY_5e7cd626a970date__d_year, DBI16Type* KEY_5e7cd626a970part__p_brand1_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size, DBDecimalType* lineorder__lo_revenue, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e7cd62b1210 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5e7cd62b1210 |= reg_date__d_datekey;
//Probe Hash table
HT_5e7cd62b1210.for_each(KEY_5e7cd62b1210, [&] __device__ (auto const SLOT_5e7cd62b1210) {
auto const [slot_first5e7cd62b1210, slot_second5e7cd62b1210] = SLOT_5e7cd62b1210;
if (!(true)) return;
uint64_t KEY_5e7cd626a970 = 0;
auto reg_date__d_year = date__d_year[tid];

KEY_5e7cd626a970 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_5e7cd62b1210[slot_second5e7cd62b1210 * 3 + 1]];
KEY_5e7cd626a970 <<= 16;
KEY_5e7cd626a970 |= reg_part__p_brand1_encoded;
//Aggregate in hashtable
auto buf_idx_5e7cd626a970 = HT_5e7cd626a970.find(KEY_5e7cd626a970)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_5e7cd62b1210[slot_second5e7cd62b1210 * 3 + 0]];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5e7cd626a970], reg_lineorder__lo_revenue);
KEY_5e7cd626a970date__d_year[buf_idx_5e7cd626a970] = reg_date__d_year;
KEY_5e7cd626a970part__p_brand1_encoded[buf_idx_5e7cd626a970] = reg_part__p_brand1_encoded;
});
}
__global__ void count_5e7cd62d4370(size_t COUNT5e7cd626a970, uint64_t* COUNT5e7cd627daf0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5e7cd626a970) return;
//Materialize count
atomicAdd((int*)COUNT5e7cd627daf0, 1);
}
__global__ void main_5e7cd62d4370(size_t COUNT5e7cd626a970, DBDecimalType* MAT5e7cd627daf0aggr0__tmp_attr0, DBI32Type* MAT5e7cd627daf0date__d_year, DBI16Type* MAT5e7cd627daf0part__p_brand1_encoded, uint64_t* MAT_IDX5e7cd627daf0, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5e7cd626a970) return;
//Materialize buffers
auto mat_idx5e7cd627daf0 = atomicAdd((int*)MAT_IDX5e7cd627daf0, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5e7cd627daf0aggr0__tmp_attr0[mat_idx5e7cd627daf0] = reg_aggr0__tmp_attr0;
auto reg_date__d_year = date__d_year[tid];
MAT5e7cd627daf0date__d_year[mat_idx5e7cd627daf0] = reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[tid];
MAT5e7cd627daf0part__p_brand1_encoded[mat_idx5e7cd627daf0] = reg_part__p_brand1_encoded;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map) {
//Materialize count
uint64_t* d_COUNT5e7cd62b1dd0;
hipMalloc(&d_COUNT5e7cd62b1dd0, sizeof(uint64_t));
hipMemset(d_COUNT5e7cd62b1dd0, 0, sizeof(uint64_t));
count_5e7cd62c1540<<<std::ceil((float)part_size/32.), 32>>>(d_COUNT5e7cd62b1dd0, d_part__p_brand1, part_size);
uint64_t COUNT5e7cd62b1dd0;
hipMemcpy(&COUNT5e7cd62b1dd0, d_COUNT5e7cd62b1dd0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5e7cd62b1dd0;
hipMalloc(&d_BUF_IDX_5e7cd62b1dd0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5e7cd62b1dd0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5e7cd62b1dd0;
hipMalloc(&d_BUF_5e7cd62b1dd0, sizeof(uint64_t) * COUNT5e7cd62b1dd0 * 1);
auto d_HT_5e7cd62b1dd0 = cuco::experimental::static_multimap{ (int)COUNT5e7cd62b1dd0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5e7cd62c1540<<<std::ceil((float)part_size/32.), 32>>>(d_BUF_5e7cd62b1dd0, d_BUF_IDX_5e7cd62b1dd0, d_HT_5e7cd62b1dd0.ref(cuco::insert), d_part__p_brand1, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT5e7cd62b7280;
hipMalloc(&d_COUNT5e7cd62b7280, sizeof(uint64_t));
hipMemset(d_COUNT5e7cd62b7280, 0, sizeof(uint64_t));
count_5e7cd62c39b0<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT5e7cd62b7280, d_supplier__s_region, supplier_size);
uint64_t COUNT5e7cd62b7280;
hipMemcpy(&COUNT5e7cd62b7280, d_COUNT5e7cd62b7280, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5e7cd62b7280;
hipMalloc(&d_BUF_IDX_5e7cd62b7280, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5e7cd62b7280, 0, sizeof(uint64_t));
uint64_t* d_BUF_5e7cd62b7280;
hipMalloc(&d_BUF_5e7cd62b7280, sizeof(uint64_t) * COUNT5e7cd62b7280 * 1);
auto d_HT_5e7cd62b7280 = cuco::experimental::static_multimap{ (int)COUNT5e7cd62b7280*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5e7cd62c39b0<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_5e7cd62b7280, d_BUF_IDX_5e7cd62b7280, d_HT_5e7cd62b7280.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5e7cd62b1210;
hipMalloc(&d_COUNT5e7cd62b1210, sizeof(uint64_t));
hipMemset(d_COUNT5e7cd62b1210, 0, sizeof(uint64_t));
count_5e7cd629a430<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_5e7cd62b1dd0, d_BUF_5e7cd62b7280, d_COUNT5e7cd62b1210, d_HT_5e7cd62b1dd0.ref(cuco::for_each), d_HT_5e7cd62b7280.ref(cuco::for_each), d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
uint64_t COUNT5e7cd62b1210;
hipMemcpy(&COUNT5e7cd62b1210, d_COUNT5e7cd62b1210, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5e7cd62b1210;
hipMalloc(&d_BUF_IDX_5e7cd62b1210, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5e7cd62b1210, 0, sizeof(uint64_t));
uint64_t* d_BUF_5e7cd62b1210;
hipMalloc(&d_BUF_5e7cd62b1210, sizeof(uint64_t) * COUNT5e7cd62b1210 * 3);
auto d_HT_5e7cd62b1210 = cuco::experimental::static_multimap{ (int)COUNT5e7cd62b1210*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5e7cd629a430<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_5e7cd62b1210, d_BUF_5e7cd62b1dd0, d_BUF_5e7cd62b7280, d_BUF_IDX_5e7cd62b1210, d_HT_5e7cd62b1210.ref(cuco::insert), d_HT_5e7cd62b1dd0.ref(cuco::for_each), d_HT_5e7cd62b7280.ref(cuco::for_each), d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
//Create aggregation hash table
auto d_HT_5e7cd626a970 = cuco::static_map{ (int)1208*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5e7cd629aa00<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_5e7cd62b1210, d_HT_5e7cd626a970.ref(cuco::insert), d_HT_5e7cd62b1210.ref(cuco::for_each), d_date__d_datekey, d_date__d_year, date_size, d_part__p_brand1_encoded);
size_t COUNT5e7cd626a970 = d_HT_5e7cd626a970.size();
thrust::device_vector<int64_t> keys_5e7cd626a970(COUNT5e7cd626a970), vals_5e7cd626a970(COUNT5e7cd626a970);
d_HT_5e7cd626a970.retrieve_all(keys_5e7cd626a970.begin(), vals_5e7cd626a970.begin());
d_HT_5e7cd626a970.clear();
int64_t* raw_keys5e7cd626a970 = thrust::raw_pointer_cast(keys_5e7cd626a970.data());
insertKeys<<<std::ceil((float)COUNT5e7cd626a970/32.), 32>>>(raw_keys5e7cd626a970, d_HT_5e7cd626a970.ref(cuco::insert), COUNT5e7cd626a970);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5e7cd626a970);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5e7cd626a970);
DBI32Type* d_KEY_5e7cd626a970date__d_year;
hipMalloc(&d_KEY_5e7cd626a970date__d_year, sizeof(DBI32Type) * COUNT5e7cd626a970);
hipMemset(d_KEY_5e7cd626a970date__d_year, 0, sizeof(DBI32Type) * COUNT5e7cd626a970);
DBI16Type* d_KEY_5e7cd626a970part__p_brand1_encoded;
hipMalloc(&d_KEY_5e7cd626a970part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5e7cd626a970);
hipMemset(d_KEY_5e7cd626a970part__p_brand1_encoded, 0, sizeof(DBI16Type) * COUNT5e7cd626a970);
main_5e7cd629aa00<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_5e7cd62b1210, d_HT_5e7cd626a970.ref(cuco::find), d_HT_5e7cd62b1210.ref(cuco::for_each), d_KEY_5e7cd626a970date__d_year, d_KEY_5e7cd626a970part__p_brand1_encoded, d_aggr0__tmp_attr0, d_date__d_datekey, d_date__d_year, date_size, d_lineorder__lo_revenue, d_part__p_brand1_encoded);
//Materialize count
uint64_t* d_COUNT5e7cd627daf0;
hipMalloc(&d_COUNT5e7cd627daf0, sizeof(uint64_t));
hipMemset(d_COUNT5e7cd627daf0, 0, sizeof(uint64_t));
count_5e7cd62d4370<<<std::ceil((float)COUNT5e7cd626a970/32.), 32>>>(COUNT5e7cd626a970, d_COUNT5e7cd627daf0);
uint64_t COUNT5e7cd627daf0;
hipMemcpy(&COUNT5e7cd627daf0, d_COUNT5e7cd627daf0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5e7cd627daf0;
hipMalloc(&d_MAT_IDX5e7cd627daf0, sizeof(uint64_t));
hipMemset(d_MAT_IDX5e7cd627daf0, 0, sizeof(uint64_t));
auto MAT5e7cd627daf0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5e7cd627daf0);
DBDecimalType* d_MAT5e7cd627daf0aggr0__tmp_attr0;
hipMalloc(&d_MAT5e7cd627daf0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5e7cd627daf0);
auto MAT5e7cd627daf0date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5e7cd627daf0);
DBI32Type* d_MAT5e7cd627daf0date__d_year;
hipMalloc(&d_MAT5e7cd627daf0date__d_year, sizeof(DBI32Type) * COUNT5e7cd627daf0);
auto MAT5e7cd627daf0part__p_brand1_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5e7cd627daf0);
DBI16Type* d_MAT5e7cd627daf0part__p_brand1_encoded;
hipMalloc(&d_MAT5e7cd627daf0part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5e7cd627daf0);
main_5e7cd62d4370<<<std::ceil((float)COUNT5e7cd626a970/32.), 32>>>(COUNT5e7cd626a970, d_MAT5e7cd627daf0aggr0__tmp_attr0, d_MAT5e7cd627daf0date__d_year, d_MAT5e7cd627daf0part__p_brand1_encoded, d_MAT_IDX5e7cd627daf0, d_aggr0__tmp_attr0, d_KEY_5e7cd626a970date__d_year, d_KEY_5e7cd626a970part__p_brand1_encoded);
hipMemcpy(MAT5e7cd627daf0aggr0__tmp_attr0, d_MAT5e7cd627daf0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5e7cd627daf0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5e7cd627daf0date__d_year, d_MAT5e7cd627daf0date__d_year, sizeof(DBI32Type) * COUNT5e7cd627daf0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5e7cd627daf0part__p_brand1_encoded, d_MAT5e7cd627daf0part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5e7cd627daf0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5e7cd627daf0; i++) { std::cout << MAT5e7cd627daf0aggr0__tmp_attr0[i] << "\t";
std::cout << MAT5e7cd627daf0date__d_year[i] << "\t";
std::cout << part__p_brand1_map[MAT5e7cd627daf0part__p_brand1_encoded[i]] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_5e7cd62b1dd0);
hipFree(d_BUF_IDX_5e7cd62b1dd0);
hipFree(d_COUNT5e7cd62b1dd0);
hipFree(d_BUF_5e7cd62b7280);
hipFree(d_BUF_IDX_5e7cd62b7280);
hipFree(d_COUNT5e7cd62b7280);
hipFree(d_BUF_5e7cd62b1210);
hipFree(d_BUF_IDX_5e7cd62b1210);
hipFree(d_COUNT5e7cd62b1210);
hipFree(d_KEY_5e7cd626a970date__d_year);
hipFree(d_KEY_5e7cd626a970part__p_brand1_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5e7cd627daf0);
hipFree(d_MAT5e7cd627daf0aggr0__tmp_attr0);
hipFree(d_MAT5e7cd627daf0date__d_year);
hipFree(d_MAT5e7cd627daf0part__p_brand1_encoded);
hipFree(d_MAT_IDX5e7cd627daf0);
free(MAT5e7cd627daf0aggr0__tmp_attr0);
free(MAT5e7cd627daf0date__d_year);
free(MAT5e7cd627daf0part__p_brand1_encoded);
}