#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_556962363290(uint64_t* COUNT5569623580d0, DBStringType* part__p_brand1, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_brand1 = part__p_brand1[tid];
if (!(evaluatePredicate(reg_part__p_brand1, "MFGR#2239", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5569623580d0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_556962363290(uint64_t* BUF_5569623580d0, uint64_t* BUF_IDX_5569623580d0, HASHTABLE_INSERT HT_5569623580d0, DBStringType* part__p_brand1, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_brand1 = part__p_brand1[tid];
if (!(evaluatePredicate(reg_part__p_brand1, "MFGR#2239", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5569623580d0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_5569623580d0 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_5569623580d0 = atomicAdd((int*)BUF_IDX_5569623580d0, 1);
HT_5569623580d0.insert(cuco::pair{KEY_5569623580d0, buf_idx_5569623580d0});
BUF_5569623580d0[buf_idx_5569623580d0 * 1 + 0] = tid;
}
__global__ void count_556962365cb0(uint64_t* COUNT556962357e00, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "EUROPE", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT556962357e00, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_556962365cb0(uint64_t* BUF_556962357e00, uint64_t* BUF_IDX_556962357e00, HASHTABLE_INSERT HT_556962357e00, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "EUROPE", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_556962357e00 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_556962357e00 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_556962357e00 = atomicAdd((int*)BUF_IDX_556962357e00, 1);
HT_556962357e00.insert(cuco::pair{KEY_556962357e00, buf_idx_556962357e00});
BUF_556962357e00[buf_idx_556962357e00 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_55696233c0e0(uint64_t* BUF_556962357e00, uint64_t* BUF_5569623580d0, uint64_t* COUNT55696233a9b0, HASHTABLE_PROBE HT_556962357e00, HASHTABLE_PROBE HT_5569623580d0, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5569623580d0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5569623580d0 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5569623580d0.for_each(KEY_5569623580d0, [&] __device__ (auto const SLOT_5569623580d0) {

auto const [slot_first5569623580d0, slot_second5569623580d0] = SLOT_5569623580d0;
if (!(true)) return;
uint64_t KEY_556962357e00 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_556962357e00 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_556962357e00.for_each(KEY_556962357e00, [&] __device__ (auto const SLOT_556962357e00) {

auto const [slot_first556962357e00, slot_second556962357e00] = SLOT_556962357e00;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT55696233a9b0, 1);
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_55696233c0e0(uint64_t* BUF_55696233a9b0, uint64_t* BUF_556962357e00, uint64_t* BUF_5569623580d0, uint64_t* BUF_IDX_55696233a9b0, HASHTABLE_INSERT HT_55696233a9b0, HASHTABLE_PROBE HT_556962357e00, HASHTABLE_PROBE HT_5569623580d0, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5569623580d0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5569623580d0 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5569623580d0.for_each(KEY_5569623580d0, [&] __device__ (auto const SLOT_5569623580d0) {
auto const [slot_first5569623580d0, slot_second5569623580d0] = SLOT_5569623580d0;
if (!(true)) return;
uint64_t KEY_556962357e00 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_556962357e00 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_556962357e00.for_each(KEY_556962357e00, [&] __device__ (auto const SLOT_556962357e00) {
auto const [slot_first556962357e00, slot_second556962357e00] = SLOT_556962357e00;
if (!(true)) return;
uint64_t KEY_55696233a9b0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_55696233a9b0 |= reg_lineorder__lo_orderdate;
// Insert hash table kernel;
auto buf_idx_55696233a9b0 = atomicAdd((int*)BUF_IDX_55696233a9b0, 1);
HT_55696233a9b0.insert(cuco::pair{KEY_55696233a9b0, buf_idx_55696233a9b0});
BUF_55696233a9b0[buf_idx_55696233a9b0 * 3 + 0] = tid;
BUF_55696233a9b0[buf_idx_55696233a9b0 * 3 + 1] = BUF_5569623580d0[slot_second5569623580d0 * 1 + 0];
BUF_55696233a9b0[buf_idx_55696233a9b0 * 3 + 2] = BUF_556962357e00[slot_second556962357e00 * 1 + 0];
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_55696233c6b0(uint64_t* BUF_55696233a9b0, HASHTABLE_INSERT HT_55696230c620, HASHTABLE_PROBE HT_55696233a9b0, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_55696233a9b0 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_55696233a9b0 |= reg_date__d_datekey;
//Probe Hash table
HT_55696233a9b0.for_each(KEY_55696233a9b0, [&] __device__ (auto const SLOT_55696233a9b0) {

auto const [slot_first55696233a9b0, slot_second55696233a9b0] = SLOT_55696233a9b0;
if (!(true)) return;
uint64_t KEY_55696230c620 = 0;
auto reg_date__d_year = date__d_year[tid];

KEY_55696230c620 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_55696233a9b0[slot_second55696233a9b0 * 3 + 1]];
KEY_55696230c620 <<= 16;
KEY_55696230c620 |= reg_part__p_brand1_encoded;
//Create aggregation hash table
HT_55696230c620.insert(cuco::pair{KEY_55696230c620, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_55696233c6b0(uint64_t* BUF_55696233a9b0, HASHTABLE_FIND HT_55696230c620, HASHTABLE_PROBE HT_55696233a9b0, DBI32Type* KEY_55696230c620date__d_year, DBI16Type* KEY_55696230c620part__p_brand1_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size, DBDecimalType* lineorder__lo_revenue, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_55696233a9b0 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_55696233a9b0 |= reg_date__d_datekey;
//Probe Hash table
HT_55696233a9b0.for_each(KEY_55696233a9b0, [&] __device__ (auto const SLOT_55696233a9b0) {
auto const [slot_first55696233a9b0, slot_second55696233a9b0] = SLOT_55696233a9b0;
if (!(true)) return;
uint64_t KEY_55696230c620 = 0;
auto reg_date__d_year = date__d_year[tid];

KEY_55696230c620 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_55696233a9b0[slot_second55696233a9b0 * 3 + 1]];
KEY_55696230c620 <<= 16;
KEY_55696230c620 |= reg_part__p_brand1_encoded;
//Aggregate in hashtable
auto buf_idx_55696230c620 = HT_55696230c620.find(KEY_55696230c620)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_55696233a9b0[slot_second55696233a9b0 * 3 + 0]];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_55696230c620], reg_lineorder__lo_revenue);
KEY_55696230c620date__d_year[buf_idx_55696230c620] = reg_date__d_year;
KEY_55696230c620part__p_brand1_encoded[buf_idx_55696230c620] = reg_part__p_brand1_encoded;
});
}
__global__ void count_556962376d60(size_t COUNT55696230c620, uint64_t* COUNT55696231f740) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT55696230c620) return;
//Materialize count
atomicAdd((int*)COUNT55696231f740, 1);
}
__global__ void main_556962376d60(size_t COUNT55696230c620, DBDecimalType* MAT55696231f740aggr0__tmp_attr0, DBI32Type* MAT55696231f740date__d_year, DBI16Type* MAT55696231f740part__p_brand1_encoded, uint64_t* MAT_IDX55696231f740, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT55696230c620) return;
//Materialize buffers
auto mat_idx55696231f740 = atomicAdd((int*)MAT_IDX55696231f740, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT55696231f740aggr0__tmp_attr0[mat_idx55696231f740] = reg_aggr0__tmp_attr0;
auto reg_date__d_year = date__d_year[tid];
MAT55696231f740date__d_year[mat_idx55696231f740] = reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[tid];
MAT55696231f740part__p_brand1_encoded[mat_idx55696231f740] = reg_part__p_brand1_encoded;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT5569623580d0;
hipMalloc(&d_COUNT5569623580d0, sizeof(uint64_t));
hipMemset(d_COUNT5569623580d0, 0, sizeof(uint64_t));
count_556962363290<<<std::ceil((float)part_size/128.), 128>>>(d_COUNT5569623580d0, d_part__p_brand1, part_size);
uint64_t COUNT5569623580d0;
hipMemcpy(&COUNT5569623580d0, d_COUNT5569623580d0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5569623580d0;
hipMalloc(&d_BUF_IDX_5569623580d0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5569623580d0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5569623580d0;
hipMalloc(&d_BUF_5569623580d0, sizeof(uint64_t) * COUNT5569623580d0 * 1);
auto d_HT_5569623580d0 = cuco::experimental::static_multimap{ (int)COUNT5569623580d0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_556962363290<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_5569623580d0, d_BUF_IDX_5569623580d0, d_HT_5569623580d0.ref(cuco::insert), d_part__p_brand1, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT556962357e00;
hipMalloc(&d_COUNT556962357e00, sizeof(uint64_t));
hipMemset(d_COUNT556962357e00, 0, sizeof(uint64_t));
count_556962365cb0<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT556962357e00, d_supplier__s_region, supplier_size);
uint64_t COUNT556962357e00;
hipMemcpy(&COUNT556962357e00, d_COUNT556962357e00, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_556962357e00;
hipMalloc(&d_BUF_IDX_556962357e00, sizeof(uint64_t));
hipMemset(d_BUF_IDX_556962357e00, 0, sizeof(uint64_t));
uint64_t* d_BUF_556962357e00;
hipMalloc(&d_BUF_556962357e00, sizeof(uint64_t) * COUNT556962357e00 * 1);
auto d_HT_556962357e00 = cuco::experimental::static_multimap{ (int)COUNT556962357e00*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_556962365cb0<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_556962357e00, d_BUF_IDX_556962357e00, d_HT_556962357e00.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT55696233a9b0;
hipMalloc(&d_COUNT55696233a9b0, sizeof(uint64_t));
hipMemset(d_COUNT55696233a9b0, 0, sizeof(uint64_t));
count_55696233c0e0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_556962357e00, d_BUF_5569623580d0, d_COUNT55696233a9b0, d_HT_556962357e00.ref(cuco::for_each), d_HT_5569623580d0.ref(cuco::for_each), d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
uint64_t COUNT55696233a9b0;
hipMemcpy(&COUNT55696233a9b0, d_COUNT55696233a9b0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_55696233a9b0;
hipMalloc(&d_BUF_IDX_55696233a9b0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_55696233a9b0, 0, sizeof(uint64_t));
uint64_t* d_BUF_55696233a9b0;
hipMalloc(&d_BUF_55696233a9b0, sizeof(uint64_t) * COUNT55696233a9b0 * 3);
auto d_HT_55696233a9b0 = cuco::experimental::static_multimap{ (int)COUNT55696233a9b0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_55696233c0e0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_55696233a9b0, d_BUF_556962357e00, d_BUF_5569623580d0, d_BUF_IDX_55696233a9b0, d_HT_55696233a9b0.ref(cuco::insert), d_HT_556962357e00.ref(cuco::for_each), d_HT_5569623580d0.ref(cuco::for_each), d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
//Create aggregation hash table
auto d_HT_55696230c620 = cuco::static_map{ (int)1208*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_55696233c6b0<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_55696233a9b0, d_HT_55696230c620.ref(cuco::insert), d_HT_55696233a9b0.ref(cuco::for_each), d_date__d_datekey, d_date__d_year, date_size, d_part__p_brand1_encoded);
size_t COUNT55696230c620 = d_HT_55696230c620.size();
thrust::device_vector<int64_t> keys_55696230c620(COUNT55696230c620), vals_55696230c620(COUNT55696230c620);
d_HT_55696230c620.retrieve_all(keys_55696230c620.begin(), vals_55696230c620.begin());
d_HT_55696230c620.clear();
int64_t* raw_keys55696230c620 = thrust::raw_pointer_cast(keys_55696230c620.data());
insertKeys<<<std::ceil((float)COUNT55696230c620/128.), 128>>>(raw_keys55696230c620, d_HT_55696230c620.ref(cuco::insert), COUNT55696230c620);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT55696230c620);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT55696230c620);
DBI32Type* d_KEY_55696230c620date__d_year;
hipMalloc(&d_KEY_55696230c620date__d_year, sizeof(DBI32Type) * COUNT55696230c620);
hipMemset(d_KEY_55696230c620date__d_year, 0, sizeof(DBI32Type) * COUNT55696230c620);
DBI16Type* d_KEY_55696230c620part__p_brand1_encoded;
hipMalloc(&d_KEY_55696230c620part__p_brand1_encoded, sizeof(DBI16Type) * COUNT55696230c620);
hipMemset(d_KEY_55696230c620part__p_brand1_encoded, 0, sizeof(DBI16Type) * COUNT55696230c620);
main_55696233c6b0<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_55696233a9b0, d_HT_55696230c620.ref(cuco::find), d_HT_55696233a9b0.ref(cuco::for_each), d_KEY_55696230c620date__d_year, d_KEY_55696230c620part__p_brand1_encoded, d_aggr0__tmp_attr0, d_date__d_datekey, d_date__d_year, date_size, d_lineorder__lo_revenue, d_part__p_brand1_encoded);
//Materialize count
uint64_t* d_COUNT55696231f740;
hipMalloc(&d_COUNT55696231f740, sizeof(uint64_t));
hipMemset(d_COUNT55696231f740, 0, sizeof(uint64_t));
count_556962376d60<<<std::ceil((float)COUNT55696230c620/128.), 128>>>(COUNT55696230c620, d_COUNT55696231f740);
uint64_t COUNT55696231f740;
hipMemcpy(&COUNT55696231f740, d_COUNT55696231f740, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX55696231f740;
hipMalloc(&d_MAT_IDX55696231f740, sizeof(uint64_t));
hipMemset(d_MAT_IDX55696231f740, 0, sizeof(uint64_t));
auto MAT55696231f740aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT55696231f740);
DBDecimalType* d_MAT55696231f740aggr0__tmp_attr0;
hipMalloc(&d_MAT55696231f740aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT55696231f740);
auto MAT55696231f740date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT55696231f740);
DBI32Type* d_MAT55696231f740date__d_year;
hipMalloc(&d_MAT55696231f740date__d_year, sizeof(DBI32Type) * COUNT55696231f740);
auto MAT55696231f740part__p_brand1_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT55696231f740);
DBI16Type* d_MAT55696231f740part__p_brand1_encoded;
hipMalloc(&d_MAT55696231f740part__p_brand1_encoded, sizeof(DBI16Type) * COUNT55696231f740);
main_556962376d60<<<std::ceil((float)COUNT55696230c620/128.), 128>>>(COUNT55696230c620, d_MAT55696231f740aggr0__tmp_attr0, d_MAT55696231f740date__d_year, d_MAT55696231f740part__p_brand1_encoded, d_MAT_IDX55696231f740, d_aggr0__tmp_attr0, d_KEY_55696230c620date__d_year, d_KEY_55696230c620part__p_brand1_encoded);
hipMemcpy(MAT55696231f740aggr0__tmp_attr0, d_MAT55696231f740aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT55696231f740, hipMemcpyDeviceToHost);
hipMemcpy(MAT55696231f740date__d_year, d_MAT55696231f740date__d_year, sizeof(DBI32Type) * COUNT55696231f740, hipMemcpyDeviceToHost);
hipMemcpy(MAT55696231f740part__p_brand1_encoded, d_MAT55696231f740part__p_brand1_encoded, sizeof(DBI16Type) * COUNT55696231f740, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT55696231f740; i++) { std::cout << "" << MAT55696231f740aggr0__tmp_attr0[i];
std::cout << "," << MAT55696231f740date__d_year[i];
std::cout << "," << part__p_brand1_map[MAT55696231f740part__p_brand1_encoded[i]];
std::cout << std::endl; }
hipFree(d_BUF_5569623580d0);
hipFree(d_BUF_IDX_5569623580d0);
hipFree(d_COUNT5569623580d0);
hipFree(d_BUF_556962357e00);
hipFree(d_BUF_IDX_556962357e00);
hipFree(d_COUNT556962357e00);
hipFree(d_BUF_55696233a9b0);
hipFree(d_BUF_IDX_55696233a9b0);
hipFree(d_COUNT55696233a9b0);
hipFree(d_KEY_55696230c620date__d_year);
hipFree(d_KEY_55696230c620part__p_brand1_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT55696231f740);
hipFree(d_MAT55696231f740aggr0__tmp_attr0);
hipFree(d_MAT55696231f740date__d_year);
hipFree(d_MAT55696231f740part__p_brand1_encoded);
hipFree(d_MAT_IDX55696231f740);
free(MAT55696231f740aggr0__tmp_attr0);
free(MAT55696231f740date__d_year);
free(MAT55696231f740part__p_brand1_encoded);
}