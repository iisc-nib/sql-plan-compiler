#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_1(uint64_t* COUNT0, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_supplier__s_region[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_region[ITEM] = supplier__s_region[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_supplier__s_region[ITEM], "AMERICA", Predicate::eq);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT0, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_supplier__s_region[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_region[ITEM] = supplier__s_region[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_supplier__s_region[ITEM], "AMERICA", Predicate::eq);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0[ITEM], buf_idx_0});
BUF_0[(buf_idx_0) * 1 + 0] = ITEM*TB + tid;
}
}
__global__ void count_3(uint64_t* COUNT2, DBStringType* customer__c_region, size_t customer_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_customer__c_region[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_region[ITEM] = customer__c_region[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_customer__c_region[ITEM], "AMERICA", Predicate::eq);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT2, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_INSERT HT_2, DBI32Type* customer__c_custkey, DBStringType* customer__c_region, size_t customer_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_customer__c_region[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_region[ITEM] = customer__c_region[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_customer__c_region[ITEM], "AMERICA", Predicate::eq);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_2 = atomicAdd((int*)BUF_IDX_2, 1);
HT_2.insert(cuco::pair{KEY_2[ITEM], buf_idx_2});
BUF_2[(buf_idx_2) * 1 + 0] = ITEM*TB + tid;
}
}
__global__ void count_5(uint64_t* COUNT4, DBI32Type* date__d_year, size_t date_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_year[ITEM] = date__d_year[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_date__d_year[ITEM], 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year[ITEM], 1998, Predicate::eq));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT4, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_INSERT HT_4, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_year[ITEM] = date__d_year[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_date__d_year[ITEM], 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year[ITEM], 1998, Predicate::eq));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_date__d_datekey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_datekey[ITEM] = date__d_datekey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_date__d_datekey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_4 = atomicAdd((int*)BUF_IDX_4, 1);
HT_4.insert(cuco::pair{KEY_4[ITEM], buf_idx_4});
BUF_4[(buf_idx_4) * 1 + 0] = ITEM*TB + tid;
}
}
__global__ void count_7(uint64_t* COUNT6, DBStringType* part__p_mfgr, size_t part_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_part__p_mfgr[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_mfgr[ITEM] = part__p_mfgr[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_part__p_mfgr[ITEM], "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr[ITEM], "MFGR#2", Predicate::eq));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT6, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_7(uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_INSERT HT_6, DBStringType* part__p_mfgr, DBI32Type* part__p_partkey, size_t part_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_part__p_mfgr[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_mfgr[ITEM] = part__p_mfgr[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_part__p_mfgr[ITEM], "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr[ITEM], "MFGR#2", Predicate::eq));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_6 = atomicAdd((int*)BUF_IDX_6, 1);
HT_6.insert(cuco::pair{KEY_6[ITEM], buf_idx_6});
BUF_6[(buf_idx_6) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_9(uint64_t* BUF_0, uint64_t* BUF_2, uint64_t* BUF_4, uint64_t* BUF_6, HASHTABLE_PROBE HT_0, HASHTABLE_PROBE HT_2, HASHTABLE_PROBE HT_4, HASHTABLE_PROBE HT_6, HASHTABLE_INSERT HT_8, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* part__p_category_encoded, DBI16Type* supplier__s_nation_encoded) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_suppkey[ITEM] = lineorder__lo_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_lineorder__lo_suppkey[ITEM];
}
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_custkey[ITEM] = lineorder__lo_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_lineorder__lo_custkey[ITEM];
}
int64_t slot_second2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_2 = HT_2.find(KEY_2[ITEM]);
if (SLOT_2 == HT_2.end()) {selection_flags[ITEM] = 0; continue;}
slot_second2[ITEM] = SLOT_2->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_orderdate[ITEM] = lineorder__lo_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_lineorder__lo_orderdate[ITEM];
}
int64_t slot_second4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0; continue;}
slot_second4[ITEM] = SLOT_4->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_partkey[ITEM] = lineorder__lo_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_lineorder__lo_partkey[ITEM];
}
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_date__d_year[ITEM] = date__d_year[BUF_4[slot_second4[ITEM] * 1 + 0]];
}
DBI16Type reg_supplier__s_nation_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_supplier__s_nation_encoded[ITEM] = supplier__s_nation_encoded[BUF_0[slot_second0[ITEM] * 1 + 0]];
}
DBI16Type reg_part__p_category_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_category_encoded[ITEM] = part__p_category_encoded[BUF_6[slot_second6[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_date__d_year[ITEM];
KEY_8[ITEM] <<= 16;
KEY_8[ITEM] |= reg_supplier__s_nation_encoded[ITEM];
KEY_8[ITEM] <<= 16;
KEY_8[ITEM] |= reg_part__p_category_encoded[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_8.insert(cuco::pair{KEY_8[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_9(uint64_t* BUF_0, uint64_t* BUF_2, uint64_t* BUF_4, uint64_t* BUF_6, HASHTABLE_PROBE HT_0, HASHTABLE_PROBE HT_2, HASHTABLE_PROBE HT_4, HASHTABLE_PROBE HT_6, HASHTABLE_FIND HT_8, DBI32Type* KEY_8date__d_year, DBI16Type* KEY_8part__p_category_encoded, DBI16Type* KEY_8supplier__s_nation_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, DBDecimalType* lineorder__lo_supplycost, size_t lineorder_size, DBI16Type* part__p_category_encoded, DBI16Type* supplier__s_nation_encoded) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_suppkey[ITEM] = lineorder__lo_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_lineorder__lo_suppkey[ITEM];
}
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_custkey[ITEM] = lineorder__lo_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_lineorder__lo_custkey[ITEM];
}
int64_t slot_second2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_2 = HT_2.find(KEY_2[ITEM]);
if (SLOT_2 == HT_2.end()) {selection_flags[ITEM] = 0; continue;}
slot_second2[ITEM] = SLOT_2->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_orderdate[ITEM] = lineorder__lo_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_lineorder__lo_orderdate[ITEM];
}
int64_t slot_second4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0; continue;}
slot_second4[ITEM] = SLOT_4->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_partkey[ITEM] = lineorder__lo_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_lineorder__lo_partkey[ITEM];
}
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_date__d_year[ITEM] = date__d_year[BUF_4[slot_second4[ITEM] * 1 + 0]];
}
DBI16Type reg_supplier__s_nation_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_supplier__s_nation_encoded[ITEM] = supplier__s_nation_encoded[BUF_0[slot_second0[ITEM] * 1 + 0]];
}
DBI16Type reg_part__p_category_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_category_encoded[ITEM] = part__p_category_encoded[BUF_6[slot_second6[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_date__d_year[ITEM];
KEY_8[ITEM] <<= 16;
KEY_8[ITEM] |= reg_supplier__s_nation_encoded[ITEM];
KEY_8[ITEM] <<= 16;
KEY_8[ITEM] |= reg_part__p_category_encoded[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineorder__lo_supplycost[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_supplycost[ITEM] = lineorder__lo_supplycost[ITEM*TB + tid];
}
DBDecimalType reg_lineorder__lo_revenue[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_revenue[ITEM] = lineorder__lo_revenue[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineorder__lo_revenue[ITEM]) - (reg_lineorder__lo_supplycost[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_8 = HT_8.find(KEY_8[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr0[buf_idx_8], reg_map0__tmp_attr1[ITEM]);
KEY_8date__d_year[buf_idx_8] = reg_date__d_year[ITEM];
KEY_8supplier__s_nation_encoded[buf_idx_8] = reg_supplier__s_nation_encoded[ITEM];
KEY_8part__p_category_encoded[buf_idx_8] = reg_part__p_category_encoded[ITEM];
}
}
__global__ void count_11(uint64_t* COUNT10, size_t COUNT8) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT8); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT10, 1);
}
}
__global__ void main_11(size_t COUNT8, DBDecimalType* MAT10aggr0__tmp_attr0, DBI32Type* MAT10date__d_year, DBI16Type* MAT10part__p_category_encoded, DBI16Type* MAT10supplier__s_nation_encoded, uint64_t* MAT_IDX10, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_category_encoded, DBI16Type* supplier__s_nation_encoded) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize buffers
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT8); ++ITEM) {
reg_date__d_year[ITEM] = date__d_year[ITEM*TB + tid];
}
DBI16Type reg_supplier__s_nation_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT8); ++ITEM) {
reg_supplier__s_nation_encoded[ITEM] = supplier__s_nation_encoded[ITEM*TB + tid];
}
DBI16Type reg_part__p_category_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT8); ++ITEM) {
reg_part__p_category_encoded[ITEM] = part__p_category_encoded[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT8); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT8); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx10 = atomicAdd((int*)MAT_IDX10, 1);
MAT10date__d_year[mat_idx10] = reg_date__d_year[ITEM];
MAT10supplier__s_nation_encoded[mat_idx10] = reg_supplier__s_nation_encoded[ITEM];
MAT10part__p_category_encoded[mat_idx10] = reg_part__p_category_encoded[ITEM];
MAT10aggr0__tmp_attr0[mat_idx10] = reg_aggr0__tmp_attr0[ITEM];
}
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
size_t used_mem = usedGpuMem();
auto start = std::chrono::high_resolution_clock::now();
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT0, d_supplier__s_region, supplier_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT2;
hipMalloc(&d_COUNT2, sizeof(uint64_t));
hipMemset(d_COUNT2, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT2, d_customer__c_region, customer_size);
uint64_t COUNT2;
hipMemcpy(&COUNT2, d_COUNT2, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 1);
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_2, d_BUF_IDX_2, d_HT_2.ref(cuco::insert), d_customer__c_custkey, d_customer__c_region, customer_size);
//Materialize count
uint64_t* d_COUNT4;
hipMalloc(&d_COUNT4, sizeof(uint64_t));
hipMemset(d_COUNT4, 0, sizeof(uint64_t));
count_5<<<std::ceil((float)date_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT4, d_date__d_year, date_size);
uint64_t COUNT4;
hipMemcpy(&COUNT4, d_COUNT4, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 1);
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)date_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_BUF_IDX_4, d_HT_4.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Materialize count
uint64_t* d_COUNT6;
hipMalloc(&d_COUNT6, sizeof(uint64_t));
hipMemset(d_COUNT6, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)part_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT6, d_part__p_mfgr, part_size);
uint64_t COUNT6;
hipMemcpy(&COUNT6, d_COUNT6, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 1);
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)part_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_6, d_BUF_IDX_6, d_HT_6.ref(cuco::insert), d_part__p_mfgr, d_part__p_partkey, part_size);
//Create aggregation hash table
auto d_HT_8 = cuco::static_map{ (int)24650*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_9<<<std::ceil((float)lineorder_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_BUF_2, d_BUF_4, d_BUF_6, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::insert), d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size, d_part__p_category_encoded, d_supplier__s_nation_encoded);
size_t COUNT8 = d_HT_8.size();
thrust::device_vector<int64_t> keys_8(COUNT8), vals_8(COUNT8);
d_HT_8.retrieve_all(keys_8.begin(), vals_8.begin());
d_HT_8.clear();
int64_t* raw_keys8 = thrust::raw_pointer_cast(keys_8.data());
insertKeys<<<std::ceil((float)COUNT8/128.), 128>>>(raw_keys8, d_HT_8.ref(cuco::insert), COUNT8);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT8);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT8);
DBI32Type* d_KEY_8date__d_year;
hipMalloc(&d_KEY_8date__d_year, sizeof(DBI32Type) * COUNT8);
hipMemset(d_KEY_8date__d_year, 0, sizeof(DBI32Type) * COUNT8);
DBI16Type* d_KEY_8supplier__s_nation_encoded;
hipMalloc(&d_KEY_8supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT8);
hipMemset(d_KEY_8supplier__s_nation_encoded, 0, sizeof(DBI16Type) * COUNT8);
DBI16Type* d_KEY_8part__p_category_encoded;
hipMalloc(&d_KEY_8part__p_category_encoded, sizeof(DBI16Type) * COUNT8);
hipMemset(d_KEY_8part__p_category_encoded, 0, sizeof(DBI16Type) * COUNT8);
main_9<<<std::ceil((float)lineorder_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_BUF_2, d_BUF_4, d_BUF_6, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::find), d_KEY_8date__d_year, d_KEY_8part__p_category_encoded, d_KEY_8supplier__s_nation_encoded, d_aggr0__tmp_attr0, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, d_lineorder__lo_supplycost, lineorder_size, d_part__p_category_encoded, d_supplier__s_nation_encoded);
//Materialize count
uint64_t* d_COUNT10;
hipMalloc(&d_COUNT10, sizeof(uint64_t));
hipMemset(d_COUNT10, 0, sizeof(uint64_t));
count_11<<<std::ceil((float)COUNT8/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT10, COUNT8);
uint64_t COUNT10;
hipMemcpy(&COUNT10, d_COUNT10, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX10;
hipMalloc(&d_MAT_IDX10, sizeof(uint64_t));
hipMemset(d_MAT_IDX10, 0, sizeof(uint64_t));
auto MAT10date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT10);
DBI32Type* d_MAT10date__d_year;
hipMalloc(&d_MAT10date__d_year, sizeof(DBI32Type) * COUNT10);
auto MAT10supplier__s_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT10);
DBI16Type* d_MAT10supplier__s_nation_encoded;
hipMalloc(&d_MAT10supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT10);
auto MAT10part__p_category_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT10);
DBI16Type* d_MAT10part__p_category_encoded;
hipMalloc(&d_MAT10part__p_category_encoded, sizeof(DBI16Type) * COUNT10);
auto MAT10aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT10);
DBDecimalType* d_MAT10aggr0__tmp_attr0;
hipMalloc(&d_MAT10aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT10);
main_11<<<std::ceil((float)COUNT8/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT8, d_MAT10aggr0__tmp_attr0, d_MAT10date__d_year, d_MAT10part__p_category_encoded, d_MAT10supplier__s_nation_encoded, d_MAT_IDX10, d_aggr0__tmp_attr0, d_KEY_8date__d_year, d_KEY_8part__p_category_encoded, d_KEY_8supplier__s_nation_encoded);
hipMemcpy(MAT10date__d_year, d_MAT10date__d_year, sizeof(DBI32Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10supplier__s_nation_encoded, d_MAT10supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10part__p_category_encoded, d_MAT10part__p_category_encoded, sizeof(DBI16Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10aggr0__tmp_attr0, d_MAT10aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT10, hipMemcpyDeviceToHost);
auto end = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < COUNT10; i++) { std::cout << "" << MAT10date__d_year[i];
std::cout << "|" << supplier__s_nation_map[MAT10supplier__s_nation_encoded[i]];
std::cout << "|" << part__p_category_map[MAT10part__p_category_encoded[i]];
std::cout << "|" << MAT10aggr0__tmp_attr0[i];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
      size_t aux_mem = usedGpuMem() - used_mem;
      std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_COUNT0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_COUNT2);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_COUNT4);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_COUNT6);
hipFree(d_KEY_8date__d_year);
hipFree(d_KEY_8part__p_category_encoded);
hipFree(d_KEY_8supplier__s_nation_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT10);
hipFree(d_MAT10aggr0__tmp_attr0);
hipFree(d_MAT10date__d_year);
hipFree(d_MAT10part__p_category_encoded);
hipFree(d_MAT10supplier__s_nation_encoded);
hipFree(d_MAT_IDX10);
free(MAT10aggr0__tmp_attr0);
free(MAT10date__d_year);
free(MAT10part__p_category_encoded);
free(MAT10supplier__s_nation_encoded);
}