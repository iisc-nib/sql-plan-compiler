#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_60b3603c3310(uint64_t* COUNT60b3603d9800, DBStringType* customer__c_city, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_city = customer__c_city[tid];
if (!((evaluatePredicate(reg_customer__c_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_customer__c_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT60b3603d9800, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_60b3603c3310(uint64_t* BUF_60b3603d9800, uint64_t* BUF_IDX_60b3603d9800, HASHTABLE_INSERT HT_60b3603d9800, DBStringType* customer__c_city, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_city = customer__c_city[tid];
if (!((evaluatePredicate(reg_customer__c_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_customer__c_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_60b3603d9800 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_60b3603d9800 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_60b3603d9800 = atomicAdd((int*)BUF_IDX_60b3603d9800, 1);
HT_60b3603d9800.insert(cuco::pair{KEY_60b3603d9800, buf_idx_60b3603d9800});
BUF_60b3603d9800[buf_idx_60b3603d9800 * 1 + 0] = tid;
}
__global__ void count_60b3603de700(uint64_t* COUNT60b3603d9b70, DBStringType* supplier__s_city, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_city = supplier__s_city[tid];
if (!((evaluatePredicate(reg_supplier__s_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_supplier__s_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT60b3603d9b70, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_60b3603de700(uint64_t* BUF_60b3603d9b70, uint64_t* BUF_IDX_60b3603d9b70, HASHTABLE_INSERT HT_60b3603d9b70, DBStringType* supplier__s_city, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_city = supplier__s_city[tid];
if (!((evaluatePredicate(reg_supplier__s_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_supplier__s_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_60b3603d9b70 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_60b3603d9b70 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_60b3603d9b70 = atomicAdd((int*)BUF_IDX_60b3603d9b70, 1);
HT_60b3603d9b70.insert(cuco::pair{KEY_60b3603d9b70, buf_idx_60b3603d9b70});
BUF_60b3603d9b70[buf_idx_60b3603d9b70 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_60b3603e0ac0(uint64_t* BUF_60b3603d9800, uint64_t* BUF_60b3603d9b70, uint64_t* COUNT60b3603e2730, HASHTABLE_PROBE HT_60b3603d9800, HASHTABLE_PROBE HT_60b3603d9b70, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_60b3603d9800 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_60b3603d9800 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_60b3603d9800.for_each(KEY_60b3603d9800, [&] __device__ (auto const SLOT_60b3603d9800) {

auto const [slot_first60b3603d9800, slot_second60b3603d9800] = SLOT_60b3603d9800;
if (!(true)) return;
uint64_t KEY_60b3603d9b70 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_60b3603d9b70 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_60b3603d9b70.for_each(KEY_60b3603d9b70, [&] __device__ (auto const SLOT_60b3603d9b70) {

auto const [slot_first60b3603d9b70, slot_second60b3603d9b70] = SLOT_60b3603d9b70;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT60b3603e2730, 1);
});
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_60b3603e0ac0(uint64_t* BUF_60b3603d9800, uint64_t* BUF_60b3603d9b70, uint64_t* BUF_60b3603e2730, uint64_t* BUF_IDX_60b3603e2730, HASHTABLE_PROBE HT_60b3603d9800, HASHTABLE_PROBE HT_60b3603d9b70, HASHTABLE_INSERT HT_60b3603e2730, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_60b3603d9800 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_60b3603d9800 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_60b3603d9800.for_each(KEY_60b3603d9800, [&] __device__ (auto const SLOT_60b3603d9800) {
auto const [slot_first60b3603d9800, slot_second60b3603d9800] = SLOT_60b3603d9800;
if (!(true)) return;
uint64_t KEY_60b3603d9b70 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_60b3603d9b70 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_60b3603d9b70.for_each(KEY_60b3603d9b70, [&] __device__ (auto const SLOT_60b3603d9b70) {
auto const [slot_first60b3603d9b70, slot_second60b3603d9b70] = SLOT_60b3603d9b70;
if (!(true)) return;
uint64_t KEY_60b3603e2730 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_60b3603e2730 |= reg_lineorder__lo_orderdate;
// Insert hash table kernel;
auto buf_idx_60b3603e2730 = atomicAdd((int*)BUF_IDX_60b3603e2730, 1);
HT_60b3603e2730.insert(cuco::pair{KEY_60b3603e2730, buf_idx_60b3603e2730});
BUF_60b3603e2730[buf_idx_60b3603e2730 * 3 + 0] = BUF_60b3603d9800[slot_second60b3603d9800 * 1 + 0];
BUF_60b3603e2730[buf_idx_60b3603e2730 * 3 + 1] = tid;
BUF_60b3603e2730[buf_idx_60b3603e2730 * 3 + 2] = BUF_60b3603d9b70[slot_second60b3603d9b70 * 1 + 0];
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_60b3603c38b0(uint64_t* BUF_60b3603e2730, HASHTABLE_INSERT HT_60b360393b20, HASHTABLE_PROBE HT_60b3603e2730, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_60b3603e2730 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_60b3603e2730 |= reg_date__d_datekey;
//Probe Hash table
HT_60b3603e2730.for_each(KEY_60b3603e2730, [&] __device__ (auto const SLOT_60b3603e2730) {

auto const [slot_first60b3603e2730, slot_second60b3603e2730] = SLOT_60b3603e2730;
if (!(true)) return;
uint64_t KEY_60b360393b20 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_60b3603e2730[slot_second60b3603e2730 * 3 + 0]];

KEY_60b360393b20 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_60b3603e2730[slot_second60b3603e2730 * 3 + 2]];
KEY_60b360393b20 <<= 16;
KEY_60b360393b20 |= reg_supplier__s_city_encoded;
KEY_60b360393b20 <<= 32;
KEY_60b360393b20 |= reg_date__d_year;
//Create aggregation hash table
HT_60b360393b20.insert(cuco::pair{KEY_60b360393b20, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_60b3603c38b0(uint64_t* BUF_60b3603e2730, HASHTABLE_FIND HT_60b360393b20, HASHTABLE_PROBE HT_60b3603e2730, DBI16Type* KEY_60b360393b20customer__c_city_encoded, DBI32Type* KEY_60b360393b20date__d_year, DBI16Type* KEY_60b360393b20supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size, DBDecimalType* lineorder__lo_revenue, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_60b3603e2730 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_60b3603e2730 |= reg_date__d_datekey;
//Probe Hash table
HT_60b3603e2730.for_each(KEY_60b3603e2730, [&] __device__ (auto const SLOT_60b3603e2730) {
auto const [slot_first60b3603e2730, slot_second60b3603e2730] = SLOT_60b3603e2730;
if (!(true)) return;
uint64_t KEY_60b360393b20 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_60b3603e2730[slot_second60b3603e2730 * 3 + 0]];

KEY_60b360393b20 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_60b3603e2730[slot_second60b3603e2730 * 3 + 2]];
KEY_60b360393b20 <<= 16;
KEY_60b360393b20 |= reg_supplier__s_city_encoded;
KEY_60b360393b20 <<= 32;
KEY_60b360393b20 |= reg_date__d_year;
//Aggregate in hashtable
auto buf_idx_60b360393b20 = HT_60b360393b20.find(KEY_60b360393b20)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_60b3603e2730[slot_second60b3603e2730 * 3 + 1]];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_60b360393b20], reg_lineorder__lo_revenue);
KEY_60b360393b20customer__c_city_encoded[buf_idx_60b360393b20] = reg_customer__c_city_encoded;
KEY_60b360393b20supplier__s_city_encoded[buf_idx_60b360393b20] = reg_supplier__s_city_encoded;
KEY_60b360393b20date__d_year[buf_idx_60b360393b20] = reg_date__d_year;
});
}
__global__ void count_60b3603fd780(size_t COUNT60b360393b20, uint64_t* COUNT60b3603a67a0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT60b360393b20) return;
//Materialize count
atomicAdd((int*)COUNT60b3603a67a0, 1);
}
__global__ void main_60b3603fd780(size_t COUNT60b360393b20, DBDecimalType* MAT60b3603a67a0aggr0__tmp_attr0, DBI16Type* MAT60b3603a67a0customer__c_city_encoded, DBI32Type* MAT60b3603a67a0date__d_year, DBI16Type* MAT60b3603a67a0supplier__s_city_encoded, uint64_t* MAT_IDX60b3603a67a0, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT60b360393b20) return;
//Materialize buffers
auto mat_idx60b3603a67a0 = atomicAdd((int*)MAT_IDX60b3603a67a0, 1);
auto reg_customer__c_city_encoded = customer__c_city_encoded[tid];
MAT60b3603a67a0customer__c_city_encoded[mat_idx60b3603a67a0] = reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[tid];
MAT60b3603a67a0supplier__s_city_encoded[mat_idx60b3603a67a0] = reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[tid];
MAT60b3603a67a0date__d_year[mat_idx60b3603a67a0] = reg_date__d_year;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT60b3603a67a0aggr0__tmp_attr0[mat_idx60b3603a67a0] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map) {
//Materialize count
uint64_t* d_COUNT60b3603d9800;
hipMalloc(&d_COUNT60b3603d9800, sizeof(uint64_t));
hipMemset(d_COUNT60b3603d9800, 0, sizeof(uint64_t));
count_60b3603c3310<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT60b3603d9800, d_customer__c_city, customer_size);
uint64_t COUNT60b3603d9800;
hipMemcpy(&COUNT60b3603d9800, d_COUNT60b3603d9800, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_60b3603d9800;
hipMalloc(&d_BUF_IDX_60b3603d9800, sizeof(uint64_t));
hipMemset(d_BUF_IDX_60b3603d9800, 0, sizeof(uint64_t));
uint64_t* d_BUF_60b3603d9800;
hipMalloc(&d_BUF_60b3603d9800, sizeof(uint64_t) * COUNT60b3603d9800 * 1);
auto d_HT_60b3603d9800 = cuco::experimental::static_multimap{ (int)COUNT60b3603d9800*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_60b3603c3310<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_60b3603d9800, d_BUF_IDX_60b3603d9800, d_HT_60b3603d9800.ref(cuco::insert), d_customer__c_city, d_customer__c_custkey, customer_size);
//Materialize count
uint64_t* d_COUNT60b3603d9b70;
hipMalloc(&d_COUNT60b3603d9b70, sizeof(uint64_t));
hipMemset(d_COUNT60b3603d9b70, 0, sizeof(uint64_t));
count_60b3603de700<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT60b3603d9b70, d_supplier__s_city, supplier_size);
uint64_t COUNT60b3603d9b70;
hipMemcpy(&COUNT60b3603d9b70, d_COUNT60b3603d9b70, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_60b3603d9b70;
hipMalloc(&d_BUF_IDX_60b3603d9b70, sizeof(uint64_t));
hipMemset(d_BUF_IDX_60b3603d9b70, 0, sizeof(uint64_t));
uint64_t* d_BUF_60b3603d9b70;
hipMalloc(&d_BUF_60b3603d9b70, sizeof(uint64_t) * COUNT60b3603d9b70 * 1);
auto d_HT_60b3603d9b70 = cuco::experimental::static_multimap{ (int)COUNT60b3603d9b70*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_60b3603de700<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_60b3603d9b70, d_BUF_IDX_60b3603d9b70, d_HT_60b3603d9b70.ref(cuco::insert), d_supplier__s_city, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT60b3603e2730;
hipMalloc(&d_COUNT60b3603e2730, sizeof(uint64_t));
hipMemset(d_COUNT60b3603e2730, 0, sizeof(uint64_t));
count_60b3603e0ac0<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_60b3603d9800, d_BUF_60b3603d9b70, d_COUNT60b3603e2730, d_HT_60b3603d9800.ref(cuco::for_each), d_HT_60b3603d9b70.ref(cuco::for_each), d_lineorder__lo_custkey, d_lineorder__lo_suppkey, lineorder_size);
uint64_t COUNT60b3603e2730;
hipMemcpy(&COUNT60b3603e2730, d_COUNT60b3603e2730, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_60b3603e2730;
hipMalloc(&d_BUF_IDX_60b3603e2730, sizeof(uint64_t));
hipMemset(d_BUF_IDX_60b3603e2730, 0, sizeof(uint64_t));
uint64_t* d_BUF_60b3603e2730;
hipMalloc(&d_BUF_60b3603e2730, sizeof(uint64_t) * COUNT60b3603e2730 * 3);
auto d_HT_60b3603e2730 = cuco::experimental::static_multimap{ (int)COUNT60b3603e2730*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_60b3603e0ac0<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_60b3603d9800, d_BUF_60b3603d9b70, d_BUF_60b3603e2730, d_BUF_IDX_60b3603e2730, d_HT_60b3603d9800.ref(cuco::for_each), d_HT_60b3603d9b70.ref(cuco::for_each), d_HT_60b3603e2730.ref(cuco::insert), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size);
//Create aggregation hash table
auto d_HT_60b360393b20 = cuco::static_map{ (int)132*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_60b3603c38b0<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_60b3603e2730, d_HT_60b360393b20.ref(cuco::insert), d_HT_60b3603e2730.ref(cuco::for_each), d_customer__c_city_encoded, d_date__d_datekey, d_date__d_year, date_size, d_supplier__s_city_encoded);
size_t COUNT60b360393b20 = d_HT_60b360393b20.size();
thrust::device_vector<int64_t> keys_60b360393b20(COUNT60b360393b20), vals_60b360393b20(COUNT60b360393b20);
d_HT_60b360393b20.retrieve_all(keys_60b360393b20.begin(), vals_60b360393b20.begin());
d_HT_60b360393b20.clear();
int64_t* raw_keys60b360393b20 = thrust::raw_pointer_cast(keys_60b360393b20.data());
insertKeys<<<std::ceil((float)COUNT60b360393b20/32.), 32>>>(raw_keys60b360393b20, d_HT_60b360393b20.ref(cuco::insert), COUNT60b360393b20);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT60b360393b20);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT60b360393b20);
DBI16Type* d_KEY_60b360393b20customer__c_city_encoded;
hipMalloc(&d_KEY_60b360393b20customer__c_city_encoded, sizeof(DBI16Type) * COUNT60b360393b20);
hipMemset(d_KEY_60b360393b20customer__c_city_encoded, 0, sizeof(DBI16Type) * COUNT60b360393b20);
DBI16Type* d_KEY_60b360393b20supplier__s_city_encoded;
hipMalloc(&d_KEY_60b360393b20supplier__s_city_encoded, sizeof(DBI16Type) * COUNT60b360393b20);
hipMemset(d_KEY_60b360393b20supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT60b360393b20);
DBI32Type* d_KEY_60b360393b20date__d_year;
hipMalloc(&d_KEY_60b360393b20date__d_year, sizeof(DBI32Type) * COUNT60b360393b20);
hipMemset(d_KEY_60b360393b20date__d_year, 0, sizeof(DBI32Type) * COUNT60b360393b20);
main_60b3603c38b0<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_60b3603e2730, d_HT_60b360393b20.ref(cuco::find), d_HT_60b3603e2730.ref(cuco::for_each), d_KEY_60b360393b20customer__c_city_encoded, d_KEY_60b360393b20date__d_year, d_KEY_60b360393b20supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_city_encoded, d_date__d_datekey, d_date__d_year, date_size, d_lineorder__lo_revenue, d_supplier__s_city_encoded);
//Materialize count
uint64_t* d_COUNT60b3603a67a0;
hipMalloc(&d_COUNT60b3603a67a0, sizeof(uint64_t));
hipMemset(d_COUNT60b3603a67a0, 0, sizeof(uint64_t));
count_60b3603fd780<<<std::ceil((float)COUNT60b360393b20/32.), 32>>>(COUNT60b360393b20, d_COUNT60b3603a67a0);
uint64_t COUNT60b3603a67a0;
hipMemcpy(&COUNT60b3603a67a0, d_COUNT60b3603a67a0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX60b3603a67a0;
hipMalloc(&d_MAT_IDX60b3603a67a0, sizeof(uint64_t));
hipMemset(d_MAT_IDX60b3603a67a0, 0, sizeof(uint64_t));
auto MAT60b3603a67a0customer__c_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT60b3603a67a0);
DBI16Type* d_MAT60b3603a67a0customer__c_city_encoded;
hipMalloc(&d_MAT60b3603a67a0customer__c_city_encoded, sizeof(DBI16Type) * COUNT60b3603a67a0);
auto MAT60b3603a67a0supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT60b3603a67a0);
DBI16Type* d_MAT60b3603a67a0supplier__s_city_encoded;
hipMalloc(&d_MAT60b3603a67a0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT60b3603a67a0);
auto MAT60b3603a67a0date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT60b3603a67a0);
DBI32Type* d_MAT60b3603a67a0date__d_year;
hipMalloc(&d_MAT60b3603a67a0date__d_year, sizeof(DBI32Type) * COUNT60b3603a67a0);
auto MAT60b3603a67a0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT60b3603a67a0);
DBDecimalType* d_MAT60b3603a67a0aggr0__tmp_attr0;
hipMalloc(&d_MAT60b3603a67a0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT60b3603a67a0);
main_60b3603fd780<<<std::ceil((float)COUNT60b360393b20/32.), 32>>>(COUNT60b360393b20, d_MAT60b3603a67a0aggr0__tmp_attr0, d_MAT60b3603a67a0customer__c_city_encoded, d_MAT60b3603a67a0date__d_year, d_MAT60b3603a67a0supplier__s_city_encoded, d_MAT_IDX60b3603a67a0, d_aggr0__tmp_attr0, d_KEY_60b360393b20customer__c_city_encoded, d_KEY_60b360393b20date__d_year, d_KEY_60b360393b20supplier__s_city_encoded);
hipMemcpy(MAT60b3603a67a0customer__c_city_encoded, d_MAT60b3603a67a0customer__c_city_encoded, sizeof(DBI16Type) * COUNT60b3603a67a0, hipMemcpyDeviceToHost);
hipMemcpy(MAT60b3603a67a0supplier__s_city_encoded, d_MAT60b3603a67a0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT60b3603a67a0, hipMemcpyDeviceToHost);
hipMemcpy(MAT60b3603a67a0date__d_year, d_MAT60b3603a67a0date__d_year, sizeof(DBI32Type) * COUNT60b3603a67a0, hipMemcpyDeviceToHost);
hipMemcpy(MAT60b3603a67a0aggr0__tmp_attr0, d_MAT60b3603a67a0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT60b3603a67a0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT60b3603a67a0; i++) { std::cout << customer__c_city_map[MAT60b3603a67a0customer__c_city_encoded[i]] << "\t";
std::cout << supplier__s_city_map[MAT60b3603a67a0supplier__s_city_encoded[i]] << "\t";
std::cout << MAT60b3603a67a0date__d_year[i] << "\t";
std::cout << MAT60b3603a67a0aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_60b3603d9800);
hipFree(d_BUF_IDX_60b3603d9800);
hipFree(d_COUNT60b3603d9800);
hipFree(d_BUF_60b3603d9b70);
hipFree(d_BUF_IDX_60b3603d9b70);
hipFree(d_COUNT60b3603d9b70);
hipFree(d_BUF_60b3603e2730);
hipFree(d_BUF_IDX_60b3603e2730);
hipFree(d_COUNT60b3603e2730);
hipFree(d_KEY_60b360393b20customer__c_city_encoded);
hipFree(d_KEY_60b360393b20date__d_year);
hipFree(d_KEY_60b360393b20supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT60b3603a67a0);
hipFree(d_MAT60b3603a67a0aggr0__tmp_attr0);
hipFree(d_MAT60b3603a67a0customer__c_city_encoded);
hipFree(d_MAT60b3603a67a0date__d_year);
hipFree(d_MAT60b3603a67a0supplier__s_city_encoded);
hipFree(d_MAT_IDX60b3603a67a0);
free(MAT60b3603a67a0aggr0__tmp_attr0);
free(MAT60b3603a67a0customer__c_city_encoded);
free(MAT60b3603a67a0date__d_year);
free(MAT60b3603a67a0supplier__s_city_encoded);
}