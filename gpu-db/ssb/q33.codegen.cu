#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5c44b26532e0(uint64_t* COUNT5c44b266cdd0, DBStringType* customer__c_city, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_city = customer__c_city[tid];
if (!((evaluatePredicate(reg_customer__c_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_customer__c_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5c44b266cdd0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5c44b26532e0(uint64_t* BUF_5c44b266cdd0, uint64_t* BUF_IDX_5c44b266cdd0, HASHTABLE_INSERT HT_5c44b266cdd0, DBStringType* customer__c_city, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_city = customer__c_city[tid];
if (!((evaluatePredicate(reg_customer__c_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_customer__c_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5c44b266cdd0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5c44b266cdd0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_5c44b266cdd0 = atomicAdd((int*)BUF_IDX_5c44b266cdd0, 1);
HT_5c44b266cdd0.insert(cuco::pair{KEY_5c44b266cdd0, buf_idx_5c44b266cdd0});
BUF_5c44b266cdd0[buf_idx_5c44b266cdd0 * 1 + 0] = tid;
}
__global__ void count_5c44b2653880(uint64_t* COUNT5c44b266cf80, DBStringType* supplier__s_city, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_city = supplier__s_city[tid];
if (!((evaluatePredicate(reg_supplier__s_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_supplier__s_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5c44b266cf80, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5c44b2653880(uint64_t* BUF_5c44b266cf80, uint64_t* BUF_IDX_5c44b266cf80, HASHTABLE_INSERT HT_5c44b266cf80, DBStringType* supplier__s_city, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_city = supplier__s_city[tid];
if (!((evaluatePredicate(reg_supplier__s_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_supplier__s_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5c44b266cf80 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5c44b266cf80 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5c44b266cf80 = atomicAdd((int*)BUF_IDX_5c44b266cf80, 1);
HT_5c44b266cf80.insert(cuco::pair{KEY_5c44b266cf80, buf_idx_5c44b266cf80});
BUF_5c44b266cf80[buf_idx_5c44b266cf80 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5c44b266f740(uint64_t* BUF_5c44b266cdd0, uint64_t* BUF_5c44b266cf80, uint64_t* COUNT5c44b266f130, HASHTABLE_PROBE HT_5c44b266cdd0, HASHTABLE_PROBE HT_5c44b266cf80, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5c44b266cdd0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5c44b266cdd0 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_5c44b266cdd0.for_each(KEY_5c44b266cdd0, [&] __device__ (auto const SLOT_5c44b266cdd0) {

auto const [slot_first5c44b266cdd0, slot_second5c44b266cdd0] = SLOT_5c44b266cdd0;
if (!(true)) return;
uint64_t KEY_5c44b266cf80 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5c44b266cf80 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5c44b266cf80.for_each(KEY_5c44b266cf80, [&] __device__ (auto const SLOT_5c44b266cf80) {

auto const [slot_first5c44b266cf80, slot_second5c44b266cf80] = SLOT_5c44b266cf80;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5c44b266f130, 1);
});
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5c44b266f740(uint64_t* BUF_5c44b266cdd0, uint64_t* BUF_5c44b266cf80, uint64_t* BUF_5c44b266f130, uint64_t* BUF_IDX_5c44b266f130, HASHTABLE_PROBE HT_5c44b266cdd0, HASHTABLE_PROBE HT_5c44b266cf80, HASHTABLE_INSERT HT_5c44b266f130, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5c44b266cdd0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5c44b266cdd0 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_5c44b266cdd0.for_each(KEY_5c44b266cdd0, [&] __device__ (auto const SLOT_5c44b266cdd0) {
auto const [slot_first5c44b266cdd0, slot_second5c44b266cdd0] = SLOT_5c44b266cdd0;
if (!(true)) return;
uint64_t KEY_5c44b266cf80 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5c44b266cf80 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5c44b266cf80.for_each(KEY_5c44b266cf80, [&] __device__ (auto const SLOT_5c44b266cf80) {
auto const [slot_first5c44b266cf80, slot_second5c44b266cf80] = SLOT_5c44b266cf80;
if (!(true)) return;
uint64_t KEY_5c44b266f130 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5c44b266f130 |= reg_lineorder__lo_orderdate;
// Insert hash table kernel;
auto buf_idx_5c44b266f130 = atomicAdd((int*)BUF_IDX_5c44b266f130, 1);
HT_5c44b266f130.insert(cuco::pair{KEY_5c44b266f130, buf_idx_5c44b266f130});
BUF_5c44b266f130[buf_idx_5c44b266f130 * 3 + 0] = BUF_5c44b266cdd0[slot_second5c44b266cdd0 * 1 + 0];
BUF_5c44b266f130[buf_idx_5c44b266f130 * 3 + 1] = tid;
BUF_5c44b266f130[buf_idx_5c44b266f130 * 3 + 2] = BUF_5c44b266cf80[slot_second5c44b266cf80 * 1 + 0];
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5c44b267ee90(uint64_t* BUF_5c44b266f130, HASHTABLE_INSERT HT_5c44b26236a0, HASHTABLE_PROBE HT_5c44b266f130, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5c44b266f130 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5c44b266f130 |= reg_date__d_datekey;
//Probe Hash table
HT_5c44b266f130.for_each(KEY_5c44b266f130, [&] __device__ (auto const SLOT_5c44b266f130) {

auto const [slot_first5c44b266f130, slot_second5c44b266f130] = SLOT_5c44b266f130;
if (!(true)) return;
uint64_t KEY_5c44b26236a0 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_5c44b266f130[slot_second5c44b266f130 * 3 + 0]];

KEY_5c44b26236a0 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_5c44b266f130[slot_second5c44b266f130 * 3 + 2]];
KEY_5c44b26236a0 <<= 16;
KEY_5c44b26236a0 |= reg_supplier__s_city_encoded;
KEY_5c44b26236a0 <<= 32;
KEY_5c44b26236a0 |= reg_date__d_year;
//Create aggregation hash table
HT_5c44b26236a0.insert(cuco::pair{KEY_5c44b26236a0, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5c44b267ee90(uint64_t* BUF_5c44b266f130, HASHTABLE_FIND HT_5c44b26236a0, HASHTABLE_PROBE HT_5c44b266f130, DBI16Type* KEY_5c44b26236a0customer__c_city_encoded, DBI32Type* KEY_5c44b26236a0date__d_year, DBI16Type* KEY_5c44b26236a0supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size, DBDecimalType* lineorder__lo_revenue, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5c44b266f130 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5c44b266f130 |= reg_date__d_datekey;
//Probe Hash table
HT_5c44b266f130.for_each(KEY_5c44b266f130, [&] __device__ (auto const SLOT_5c44b266f130) {
auto const [slot_first5c44b266f130, slot_second5c44b266f130] = SLOT_5c44b266f130;
if (!(true)) return;
uint64_t KEY_5c44b26236a0 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_5c44b266f130[slot_second5c44b266f130 * 3 + 0]];

KEY_5c44b26236a0 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_5c44b266f130[slot_second5c44b266f130 * 3 + 2]];
KEY_5c44b26236a0 <<= 16;
KEY_5c44b26236a0 |= reg_supplier__s_city_encoded;
KEY_5c44b26236a0 <<= 32;
KEY_5c44b26236a0 |= reg_date__d_year;
//Aggregate in hashtable
auto buf_idx_5c44b26236a0 = HT_5c44b26236a0.find(KEY_5c44b26236a0)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_5c44b266f130[slot_second5c44b266f130 * 3 + 1]];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5c44b26236a0], reg_lineorder__lo_revenue);
KEY_5c44b26236a0customer__c_city_encoded[buf_idx_5c44b26236a0] = reg_customer__c_city_encoded;
KEY_5c44b26236a0supplier__s_city_encoded[buf_idx_5c44b26236a0] = reg_supplier__s_city_encoded;
KEY_5c44b26236a0date__d_year[buf_idx_5c44b26236a0] = reg_date__d_year;
});
}
__global__ void count_5c44b268dc50(size_t COUNT5c44b26236a0, uint64_t* COUNT5c44b2636790) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5c44b26236a0) return;
//Materialize count
atomicAdd((int*)COUNT5c44b2636790, 1);
}
__global__ void main_5c44b268dc50(size_t COUNT5c44b26236a0, DBDecimalType* MAT5c44b2636790aggr0__tmp_attr0, DBI16Type* MAT5c44b2636790customer__c_city_encoded, DBI32Type* MAT5c44b2636790date__d_year, DBI16Type* MAT5c44b2636790supplier__s_city_encoded, uint64_t* MAT_IDX5c44b2636790, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5c44b26236a0) return;
//Materialize buffers
auto mat_idx5c44b2636790 = atomicAdd((int*)MAT_IDX5c44b2636790, 1);
auto reg_customer__c_city_encoded = customer__c_city_encoded[tid];
MAT5c44b2636790customer__c_city_encoded[mat_idx5c44b2636790] = reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[tid];
MAT5c44b2636790supplier__s_city_encoded[mat_idx5c44b2636790] = reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[tid];
MAT5c44b2636790date__d_year[mat_idx5c44b2636790] = reg_date__d_year;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5c44b2636790aggr0__tmp_attr0[mat_idx5c44b2636790] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT5c44b266cdd0;
hipMalloc(&d_COUNT5c44b266cdd0, sizeof(uint64_t));
hipMemset(d_COUNT5c44b266cdd0, 0, sizeof(uint64_t));
count_5c44b26532e0<<<std::ceil((float)customer_size/128.), 128>>>(d_COUNT5c44b266cdd0, d_customer__c_city, customer_size);
uint64_t COUNT5c44b266cdd0;
hipMemcpy(&COUNT5c44b266cdd0, d_COUNT5c44b266cdd0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5c44b266cdd0;
hipMalloc(&d_BUF_IDX_5c44b266cdd0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5c44b266cdd0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5c44b266cdd0;
hipMalloc(&d_BUF_5c44b266cdd0, sizeof(uint64_t) * COUNT5c44b266cdd0 * 1);
auto d_HT_5c44b266cdd0 = cuco::experimental::static_multimap{ (int)COUNT5c44b266cdd0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5c44b26532e0<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_5c44b266cdd0, d_BUF_IDX_5c44b266cdd0, d_HT_5c44b266cdd0.ref(cuco::insert), d_customer__c_city, d_customer__c_custkey, customer_size);
//Materialize count
uint64_t* d_COUNT5c44b266cf80;
hipMalloc(&d_COUNT5c44b266cf80, sizeof(uint64_t));
hipMemset(d_COUNT5c44b266cf80, 0, sizeof(uint64_t));
count_5c44b2653880<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT5c44b266cf80, d_supplier__s_city, supplier_size);
uint64_t COUNT5c44b266cf80;
hipMemcpy(&COUNT5c44b266cf80, d_COUNT5c44b266cf80, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5c44b266cf80;
hipMalloc(&d_BUF_IDX_5c44b266cf80, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5c44b266cf80, 0, sizeof(uint64_t));
uint64_t* d_BUF_5c44b266cf80;
hipMalloc(&d_BUF_5c44b266cf80, sizeof(uint64_t) * COUNT5c44b266cf80 * 1);
auto d_HT_5c44b266cf80 = cuco::experimental::static_multimap{ (int)COUNT5c44b266cf80*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5c44b2653880<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_5c44b266cf80, d_BUF_IDX_5c44b266cf80, d_HT_5c44b266cf80.ref(cuco::insert), d_supplier__s_city, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5c44b266f130;
hipMalloc(&d_COUNT5c44b266f130, sizeof(uint64_t));
hipMemset(d_COUNT5c44b266f130, 0, sizeof(uint64_t));
count_5c44b266f740<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5c44b266cdd0, d_BUF_5c44b266cf80, d_COUNT5c44b266f130, d_HT_5c44b266cdd0.ref(cuco::for_each), d_HT_5c44b266cf80.ref(cuco::for_each), d_lineorder__lo_custkey, d_lineorder__lo_suppkey, lineorder_size);
uint64_t COUNT5c44b266f130;
hipMemcpy(&COUNT5c44b266f130, d_COUNT5c44b266f130, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5c44b266f130;
hipMalloc(&d_BUF_IDX_5c44b266f130, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5c44b266f130, 0, sizeof(uint64_t));
uint64_t* d_BUF_5c44b266f130;
hipMalloc(&d_BUF_5c44b266f130, sizeof(uint64_t) * COUNT5c44b266f130 * 3);
auto d_HT_5c44b266f130 = cuco::experimental::static_multimap{ (int)COUNT5c44b266f130*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5c44b266f740<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5c44b266cdd0, d_BUF_5c44b266cf80, d_BUF_5c44b266f130, d_BUF_IDX_5c44b266f130, d_HT_5c44b266cdd0.ref(cuco::for_each), d_HT_5c44b266cf80.ref(cuco::for_each), d_HT_5c44b266f130.ref(cuco::insert), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size);
//Create aggregation hash table
auto d_HT_5c44b26236a0 = cuco::static_map{ (int)132*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5c44b267ee90<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_5c44b266f130, d_HT_5c44b26236a0.ref(cuco::insert), d_HT_5c44b266f130.ref(cuco::for_each), d_customer__c_city_encoded, d_date__d_datekey, d_date__d_year, date_size, d_supplier__s_city_encoded);
size_t COUNT5c44b26236a0 = d_HT_5c44b26236a0.size();
thrust::device_vector<int64_t> keys_5c44b26236a0(COUNT5c44b26236a0), vals_5c44b26236a0(COUNT5c44b26236a0);
d_HT_5c44b26236a0.retrieve_all(keys_5c44b26236a0.begin(), vals_5c44b26236a0.begin());
d_HT_5c44b26236a0.clear();
int64_t* raw_keys5c44b26236a0 = thrust::raw_pointer_cast(keys_5c44b26236a0.data());
insertKeys<<<std::ceil((float)COUNT5c44b26236a0/128.), 128>>>(raw_keys5c44b26236a0, d_HT_5c44b26236a0.ref(cuco::insert), COUNT5c44b26236a0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5c44b26236a0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5c44b26236a0);
DBI16Type* d_KEY_5c44b26236a0customer__c_city_encoded;
hipMalloc(&d_KEY_5c44b26236a0customer__c_city_encoded, sizeof(DBI16Type) * COUNT5c44b26236a0);
hipMemset(d_KEY_5c44b26236a0customer__c_city_encoded, 0, sizeof(DBI16Type) * COUNT5c44b26236a0);
DBI16Type* d_KEY_5c44b26236a0supplier__s_city_encoded;
hipMalloc(&d_KEY_5c44b26236a0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5c44b26236a0);
hipMemset(d_KEY_5c44b26236a0supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT5c44b26236a0);
DBI32Type* d_KEY_5c44b26236a0date__d_year;
hipMalloc(&d_KEY_5c44b26236a0date__d_year, sizeof(DBI32Type) * COUNT5c44b26236a0);
hipMemset(d_KEY_5c44b26236a0date__d_year, 0, sizeof(DBI32Type) * COUNT5c44b26236a0);
main_5c44b267ee90<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_5c44b266f130, d_HT_5c44b26236a0.ref(cuco::find), d_HT_5c44b266f130.ref(cuco::for_each), d_KEY_5c44b26236a0customer__c_city_encoded, d_KEY_5c44b26236a0date__d_year, d_KEY_5c44b26236a0supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_city_encoded, d_date__d_datekey, d_date__d_year, date_size, d_lineorder__lo_revenue, d_supplier__s_city_encoded);
//Materialize count
uint64_t* d_COUNT5c44b2636790;
hipMalloc(&d_COUNT5c44b2636790, sizeof(uint64_t));
hipMemset(d_COUNT5c44b2636790, 0, sizeof(uint64_t));
count_5c44b268dc50<<<std::ceil((float)COUNT5c44b26236a0/128.), 128>>>(COUNT5c44b26236a0, d_COUNT5c44b2636790);
uint64_t COUNT5c44b2636790;
hipMemcpy(&COUNT5c44b2636790, d_COUNT5c44b2636790, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5c44b2636790;
hipMalloc(&d_MAT_IDX5c44b2636790, sizeof(uint64_t));
hipMemset(d_MAT_IDX5c44b2636790, 0, sizeof(uint64_t));
auto MAT5c44b2636790customer__c_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5c44b2636790);
DBI16Type* d_MAT5c44b2636790customer__c_city_encoded;
hipMalloc(&d_MAT5c44b2636790customer__c_city_encoded, sizeof(DBI16Type) * COUNT5c44b2636790);
auto MAT5c44b2636790supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5c44b2636790);
DBI16Type* d_MAT5c44b2636790supplier__s_city_encoded;
hipMalloc(&d_MAT5c44b2636790supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5c44b2636790);
auto MAT5c44b2636790date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5c44b2636790);
DBI32Type* d_MAT5c44b2636790date__d_year;
hipMalloc(&d_MAT5c44b2636790date__d_year, sizeof(DBI32Type) * COUNT5c44b2636790);
auto MAT5c44b2636790aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5c44b2636790);
DBDecimalType* d_MAT5c44b2636790aggr0__tmp_attr0;
hipMalloc(&d_MAT5c44b2636790aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5c44b2636790);
main_5c44b268dc50<<<std::ceil((float)COUNT5c44b26236a0/128.), 128>>>(COUNT5c44b26236a0, d_MAT5c44b2636790aggr0__tmp_attr0, d_MAT5c44b2636790customer__c_city_encoded, d_MAT5c44b2636790date__d_year, d_MAT5c44b2636790supplier__s_city_encoded, d_MAT_IDX5c44b2636790, d_aggr0__tmp_attr0, d_KEY_5c44b26236a0customer__c_city_encoded, d_KEY_5c44b26236a0date__d_year, d_KEY_5c44b26236a0supplier__s_city_encoded);
hipMemcpy(MAT5c44b2636790customer__c_city_encoded, d_MAT5c44b2636790customer__c_city_encoded, sizeof(DBI16Type) * COUNT5c44b2636790, hipMemcpyDeviceToHost);
hipMemcpy(MAT5c44b2636790supplier__s_city_encoded, d_MAT5c44b2636790supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5c44b2636790, hipMemcpyDeviceToHost);
hipMemcpy(MAT5c44b2636790date__d_year, d_MAT5c44b2636790date__d_year, sizeof(DBI32Type) * COUNT5c44b2636790, hipMemcpyDeviceToHost);
hipMemcpy(MAT5c44b2636790aggr0__tmp_attr0, d_MAT5c44b2636790aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5c44b2636790, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5c44b2636790; i++) { std::cout << "" << customer__c_city_map[MAT5c44b2636790customer__c_city_encoded[i]];
std::cout << "," << supplier__s_city_map[MAT5c44b2636790supplier__s_city_encoded[i]];
std::cout << "," << MAT5c44b2636790date__d_year[i];
std::cout << "," << MAT5c44b2636790aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_5c44b266cdd0);
hipFree(d_BUF_IDX_5c44b266cdd0);
hipFree(d_COUNT5c44b266cdd0);
hipFree(d_BUF_5c44b266cf80);
hipFree(d_BUF_IDX_5c44b266cf80);
hipFree(d_COUNT5c44b266cf80);
hipFree(d_BUF_5c44b266f130);
hipFree(d_BUF_IDX_5c44b266f130);
hipFree(d_COUNT5c44b266f130);
hipFree(d_KEY_5c44b26236a0customer__c_city_encoded);
hipFree(d_KEY_5c44b26236a0date__d_year);
hipFree(d_KEY_5c44b26236a0supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5c44b2636790);
hipFree(d_MAT5c44b2636790aggr0__tmp_attr0);
hipFree(d_MAT5c44b2636790customer__c_city_encoded);
hipFree(d_MAT5c44b2636790date__d_year);
hipFree(d_MAT5c44b2636790supplier__s_city_encoded);
hipFree(d_MAT_IDX5c44b2636790);
free(MAT5c44b2636790aggr0__tmp_attr0);
free(MAT5c44b2636790customer__c_city_encoded);
free(MAT5c44b2636790date__d_year);
free(MAT5c44b2636790supplier__s_city_encoded);
}