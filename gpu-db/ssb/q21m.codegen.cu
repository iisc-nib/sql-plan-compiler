#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5e61cd9a89e0(uint64_t* COUNT5e61cd998030, DBStringType* part__p_category, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_category = part__p_category[tid];
if (!(evaluatePredicate(reg_part__p_category, "MFGR#12", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5e61cd998030, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5e61cd9a89e0(uint64_t* BUF_5e61cd998030, uint64_t* BUF_IDX_5e61cd998030, HASHTABLE_INSERT HT_5e61cd998030, DBStringType* part__p_category, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_category = part__p_category[tid];
if (!(evaluatePredicate(reg_part__p_category, "MFGR#12", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e61cd998030 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_5e61cd998030 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_5e61cd998030 = atomicAdd((int*)BUF_IDX_5e61cd998030, 1);
HT_5e61cd998030.insert(cuco::pair{KEY_5e61cd998030, buf_idx_5e61cd998030});
BUF_5e61cd998030[buf_idx_5e61cd998030 * 1 + 0] = tid;
}
__global__ void count_5e61cd9ab480(uint64_t* COUNT5e61cd99dc00, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5e61cd99dc00, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5e61cd9ab480(uint64_t* BUF_5e61cd99dc00, uint64_t* BUF_IDX_5e61cd99dc00, HASHTABLE_INSERT HT_5e61cd99dc00, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e61cd99dc00 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5e61cd99dc00 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5e61cd99dc00 = atomicAdd((int*)BUF_IDX_5e61cd99dc00, 1);
HT_5e61cd99dc00.insert(cuco::pair{KEY_5e61cd99dc00, buf_idx_5e61cd99dc00});
BUF_5e61cd99dc00[buf_idx_5e61cd99dc00 * 1 + 0] = tid;
}
__global__ void count_5e61cd981b70(uint64_t* COUNT5e61cd998400, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5e61cd998400, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5e61cd981b70(uint64_t* BUF_5e61cd998400, uint64_t* BUF_IDX_5e61cd998400, HASHTABLE_INSERT HT_5e61cd998400, DBI32Type* date__d_datekey, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e61cd998400 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5e61cd998400 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_5e61cd998400 = atomicAdd((int*)BUF_IDX_5e61cd998400, 1);
HT_5e61cd998400.insert(cuco::pair{KEY_5e61cd998400, buf_idx_5e61cd998400});
BUF_5e61cd998400[buf_idx_5e61cd998400 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5e61cd9815a0(uint64_t* BUF_5e61cd998030, uint64_t* BUF_5e61cd998400, uint64_t* BUF_5e61cd99dc00, HASHTABLE_INSERT HT_5e61cd951f30, HASHTABLE_PROBE HT_5e61cd998030, HASHTABLE_PROBE HT_5e61cd998400, HASHTABLE_PROBE HT_5e61cd99dc00, DBI32Type* date__d_year, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e61cd998030 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5e61cd998030 |= reg_lineorder__lo_partkey;
//Probe Hash table
auto SLOT_5e61cd998030 = HT_5e61cd998030.find(KEY_5e61cd998030);
if (SLOT_5e61cd998030 == HT_5e61cd998030.end()) return;
if (!(true)) return;
uint64_t KEY_5e61cd99dc00 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5e61cd99dc00 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_5e61cd99dc00 = HT_5e61cd99dc00.find(KEY_5e61cd99dc00);
if (SLOT_5e61cd99dc00 == HT_5e61cd99dc00.end()) return;
if (!(true)) return;
uint64_t KEY_5e61cd998400 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5e61cd998400 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_5e61cd998400 = HT_5e61cd998400.find(KEY_5e61cd998400);
if (SLOT_5e61cd998400 == HT_5e61cd998400.end()) return;
if (!(true)) return;
uint64_t KEY_5e61cd951f30 = 0;
auto reg_date__d_year = date__d_year[BUF_5e61cd998400[SLOT_5e61cd998400->second * 1 + 0]];

KEY_5e61cd951f30 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_5e61cd998030[SLOT_5e61cd998030->second * 1 + 0]];
KEY_5e61cd951f30 <<= 16;
KEY_5e61cd951f30 |= reg_part__p_brand1_encoded;
//Create aggregation hash table
HT_5e61cd951f30.insert(cuco::pair{KEY_5e61cd951f30, 1});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5e61cd9815a0(uint64_t* BUF_5e61cd998030, uint64_t* BUF_5e61cd998400, uint64_t* BUF_5e61cd99dc00, HASHTABLE_FIND HT_5e61cd951f30, HASHTABLE_PROBE HT_5e61cd998030, HASHTABLE_PROBE HT_5e61cd998400, HASHTABLE_PROBE HT_5e61cd99dc00, DBI32Type* KEY_5e61cd951f30date__d_year, DBI16Type* KEY_5e61cd951f30part__p_brand1_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e61cd998030 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5e61cd998030 |= reg_lineorder__lo_partkey;
//Probe Hash table
auto SLOT_5e61cd998030 = HT_5e61cd998030.find(KEY_5e61cd998030);
if (SLOT_5e61cd998030 == HT_5e61cd998030.end()) return;
if (!(true)) return;
uint64_t KEY_5e61cd99dc00 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5e61cd99dc00 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_5e61cd99dc00 = HT_5e61cd99dc00.find(KEY_5e61cd99dc00);
if (SLOT_5e61cd99dc00 == HT_5e61cd99dc00.end()) return;
if (!(true)) return;
uint64_t KEY_5e61cd998400 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5e61cd998400 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_5e61cd998400 = HT_5e61cd998400.find(KEY_5e61cd998400);
if (SLOT_5e61cd998400 == HT_5e61cd998400.end()) return;
if (!(true)) return;
uint64_t KEY_5e61cd951f30 = 0;
auto reg_date__d_year = date__d_year[BUF_5e61cd998400[SLOT_5e61cd998400->second * 1 + 0]];

KEY_5e61cd951f30 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_5e61cd998030[SLOT_5e61cd998030->second * 1 + 0]];
KEY_5e61cd951f30 <<= 16;
KEY_5e61cd951f30 |= reg_part__p_brand1_encoded;
//Aggregate in hashtable
auto buf_idx_5e61cd951f30 = HT_5e61cd951f30.find(KEY_5e61cd951f30)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5e61cd951f30], reg_lineorder__lo_revenue);
KEY_5e61cd951f30date__d_year[buf_idx_5e61cd951f30] = reg_date__d_year;
KEY_5e61cd951f30part__p_brand1_encoded[buf_idx_5e61cd951f30] = reg_part__p_brand1_encoded;
}
__global__ void count_5e61cd9ba960(size_t COUNT5e61cd951f30, uint64_t* COUNT5e61cd965460) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5e61cd951f30) return;
//Materialize count
atomicAdd((int*)COUNT5e61cd965460, 1);
}
__global__ void main_5e61cd9ba960(size_t COUNT5e61cd951f30, DBDecimalType* MAT5e61cd965460aggr0__tmp_attr0, DBI32Type* MAT5e61cd965460date__d_year, DBI16Type* MAT5e61cd965460part__p_brand1_encoded, uint64_t* MAT_IDX5e61cd965460, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5e61cd951f30) return;
//Materialize buffers
auto mat_idx5e61cd965460 = atomicAdd((int*)MAT_IDX5e61cd965460, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5e61cd965460aggr0__tmp_attr0[mat_idx5e61cd965460] = reg_aggr0__tmp_attr0;
auto reg_date__d_year = date__d_year[tid];
MAT5e61cd965460date__d_year[mat_idx5e61cd965460] = reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[tid];
MAT5e61cd965460part__p_brand1_encoded[mat_idx5e61cd965460] = reg_part__p_brand1_encoded;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT5e61cd998030;
hipMalloc(&d_COUNT5e61cd998030, sizeof(uint64_t));
hipMemset(d_COUNT5e61cd998030, 0, sizeof(uint64_t));
count_5e61cd9a89e0<<<std::ceil((float)part_size/128.), 128>>>(d_COUNT5e61cd998030, d_part__p_category, part_size);
uint64_t COUNT5e61cd998030;
hipMemcpy(&COUNT5e61cd998030, d_COUNT5e61cd998030, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5e61cd998030;
hipMalloc(&d_BUF_IDX_5e61cd998030, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5e61cd998030, 0, sizeof(uint64_t));
uint64_t* d_BUF_5e61cd998030;
hipMalloc(&d_BUF_5e61cd998030, sizeof(uint64_t) * COUNT5e61cd998030 * 1);
auto d_HT_5e61cd998030 = cuco::static_map{ (int)COUNT5e61cd998030*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5e61cd9a89e0<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_5e61cd998030, d_BUF_IDX_5e61cd998030, d_HT_5e61cd998030.ref(cuco::insert), d_part__p_category, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT5e61cd99dc00;
hipMalloc(&d_COUNT5e61cd99dc00, sizeof(uint64_t));
hipMemset(d_COUNT5e61cd99dc00, 0, sizeof(uint64_t));
count_5e61cd9ab480<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT5e61cd99dc00, d_supplier__s_region, supplier_size);
uint64_t COUNT5e61cd99dc00;
hipMemcpy(&COUNT5e61cd99dc00, d_COUNT5e61cd99dc00, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5e61cd99dc00;
hipMalloc(&d_BUF_IDX_5e61cd99dc00, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5e61cd99dc00, 0, sizeof(uint64_t));
uint64_t* d_BUF_5e61cd99dc00;
hipMalloc(&d_BUF_5e61cd99dc00, sizeof(uint64_t) * COUNT5e61cd99dc00 * 1);
auto d_HT_5e61cd99dc00 = cuco::static_map{ (int)COUNT5e61cd99dc00*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5e61cd9ab480<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_5e61cd99dc00, d_BUF_IDX_5e61cd99dc00, d_HT_5e61cd99dc00.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5e61cd998400;
hipMalloc(&d_COUNT5e61cd998400, sizeof(uint64_t));
hipMemset(d_COUNT5e61cd998400, 0, sizeof(uint64_t));
count_5e61cd981b70<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT5e61cd998400, date_size);
uint64_t COUNT5e61cd998400;
hipMemcpy(&COUNT5e61cd998400, d_COUNT5e61cd998400, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5e61cd998400;
hipMalloc(&d_BUF_IDX_5e61cd998400, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5e61cd998400, 0, sizeof(uint64_t));
uint64_t* d_BUF_5e61cd998400;
hipMalloc(&d_BUF_5e61cd998400, sizeof(uint64_t) * COUNT5e61cd998400 * 1);
auto d_HT_5e61cd998400 = cuco::static_map{ (int)COUNT5e61cd998400*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5e61cd981b70<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_5e61cd998400, d_BUF_IDX_5e61cd998400, d_HT_5e61cd998400.ref(cuco::insert), d_date__d_datekey, date_size);
//Create aggregation hash table
auto d_HT_5e61cd951f30 = cuco::static_map{ (int)52974*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5e61cd9815a0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5e61cd998030, d_BUF_5e61cd998400, d_BUF_5e61cd99dc00, d_HT_5e61cd951f30.ref(cuco::insert), d_HT_5e61cd998030.ref(cuco::find), d_HT_5e61cd998400.ref(cuco::find), d_HT_5e61cd99dc00.ref(cuco::find), d_date__d_year, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size, d_part__p_brand1_encoded);
size_t COUNT5e61cd951f30 = d_HT_5e61cd951f30.size();
thrust::device_vector<int64_t> keys_5e61cd951f30(COUNT5e61cd951f30), vals_5e61cd951f30(COUNT5e61cd951f30);
d_HT_5e61cd951f30.retrieve_all(keys_5e61cd951f30.begin(), vals_5e61cd951f30.begin());
d_HT_5e61cd951f30.clear();
int64_t* raw_keys5e61cd951f30 = thrust::raw_pointer_cast(keys_5e61cd951f30.data());
insertKeys<<<std::ceil((float)COUNT5e61cd951f30/128.), 128>>>(raw_keys5e61cd951f30, d_HT_5e61cd951f30.ref(cuco::insert), COUNT5e61cd951f30);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5e61cd951f30);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5e61cd951f30);
DBI32Type* d_KEY_5e61cd951f30date__d_year;
hipMalloc(&d_KEY_5e61cd951f30date__d_year, sizeof(DBI32Type) * COUNT5e61cd951f30);
hipMemset(d_KEY_5e61cd951f30date__d_year, 0, sizeof(DBI32Type) * COUNT5e61cd951f30);
DBI16Type* d_KEY_5e61cd951f30part__p_brand1_encoded;
hipMalloc(&d_KEY_5e61cd951f30part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5e61cd951f30);
hipMemset(d_KEY_5e61cd951f30part__p_brand1_encoded, 0, sizeof(DBI16Type) * COUNT5e61cd951f30);
main_5e61cd9815a0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5e61cd998030, d_BUF_5e61cd998400, d_BUF_5e61cd99dc00, d_HT_5e61cd951f30.ref(cuco::find), d_HT_5e61cd998030.ref(cuco::find), d_HT_5e61cd998400.ref(cuco::find), d_HT_5e61cd99dc00.ref(cuco::find), d_KEY_5e61cd951f30date__d_year, d_KEY_5e61cd951f30part__p_brand1_encoded, d_aggr0__tmp_attr0, d_date__d_year, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_part__p_brand1_encoded);
//Materialize count
uint64_t* d_COUNT5e61cd965460;
hipMalloc(&d_COUNT5e61cd965460, sizeof(uint64_t));
hipMemset(d_COUNT5e61cd965460, 0, sizeof(uint64_t));
count_5e61cd9ba960<<<std::ceil((float)COUNT5e61cd951f30/128.), 128>>>(COUNT5e61cd951f30, d_COUNT5e61cd965460);
uint64_t COUNT5e61cd965460;
hipMemcpy(&COUNT5e61cd965460, d_COUNT5e61cd965460, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5e61cd965460;
hipMalloc(&d_MAT_IDX5e61cd965460, sizeof(uint64_t));
hipMemset(d_MAT_IDX5e61cd965460, 0, sizeof(uint64_t));
auto MAT5e61cd965460aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5e61cd965460);
DBDecimalType* d_MAT5e61cd965460aggr0__tmp_attr0;
hipMalloc(&d_MAT5e61cd965460aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5e61cd965460);
auto MAT5e61cd965460date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5e61cd965460);
DBI32Type* d_MAT5e61cd965460date__d_year;
hipMalloc(&d_MAT5e61cd965460date__d_year, sizeof(DBI32Type) * COUNT5e61cd965460);
auto MAT5e61cd965460part__p_brand1_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5e61cd965460);
DBI16Type* d_MAT5e61cd965460part__p_brand1_encoded;
hipMalloc(&d_MAT5e61cd965460part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5e61cd965460);
main_5e61cd9ba960<<<std::ceil((float)COUNT5e61cd951f30/128.), 128>>>(COUNT5e61cd951f30, d_MAT5e61cd965460aggr0__tmp_attr0, d_MAT5e61cd965460date__d_year, d_MAT5e61cd965460part__p_brand1_encoded, d_MAT_IDX5e61cd965460, d_aggr0__tmp_attr0, d_KEY_5e61cd951f30date__d_year, d_KEY_5e61cd951f30part__p_brand1_encoded);
hipMemcpy(MAT5e61cd965460aggr0__tmp_attr0, d_MAT5e61cd965460aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5e61cd965460, hipMemcpyDeviceToHost);
hipMemcpy(MAT5e61cd965460date__d_year, d_MAT5e61cd965460date__d_year, sizeof(DBI32Type) * COUNT5e61cd965460, hipMemcpyDeviceToHost);
hipMemcpy(MAT5e61cd965460part__p_brand1_encoded, d_MAT5e61cd965460part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5e61cd965460, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5e61cd965460; i++) { std::cout << "" << MAT5e61cd965460aggr0__tmp_attr0[i];
std::cout << "," << MAT5e61cd965460date__d_year[i];
std::cout << "," << part__p_brand1_map[MAT5e61cd965460part__p_brand1_encoded[i]];
std::cout << std::endl; }
hipFree(d_BUF_5e61cd998030);
hipFree(d_BUF_IDX_5e61cd998030);
hipFree(d_COUNT5e61cd998030);
hipFree(d_BUF_5e61cd99dc00);
hipFree(d_BUF_IDX_5e61cd99dc00);
hipFree(d_COUNT5e61cd99dc00);
hipFree(d_BUF_5e61cd998400);
hipFree(d_BUF_IDX_5e61cd998400);
hipFree(d_COUNT5e61cd998400);
hipFree(d_KEY_5e61cd951f30date__d_year);
hipFree(d_KEY_5e61cd951f30part__p_brand1_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5e61cd965460);
hipFree(d_MAT5e61cd965460aggr0__tmp_attr0);
hipFree(d_MAT5e61cd965460date__d_year);
hipFree(d_MAT5e61cd965460part__p_brand1_encoded);
hipFree(d_MAT_IDX5e61cd965460);
free(MAT5e61cd965460aggr0__tmp_attr0);
free(MAT5e61cd965460date__d_year);
free(MAT5e61cd965460part__p_brand1_encoded);
}