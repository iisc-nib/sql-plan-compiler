#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_1(uint64_t* COUNT0, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBI32Type* customer__c_custkey, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
auto reg_customer__c_custkey = customer__c_custkey[tid];
if (!(evaluatePredicate(reg_customer__c_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_0 = 0;

KEY_0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0, buf_idx_0});
BUF_0[buf_idx_0 * 1 + 0] = tid;
}
__global__ void count_3(uint64_t* COUNT2, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT2, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_INSERT HT_2, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_2 = 0;

KEY_2 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_2 = atomicAdd((int*)BUF_IDX_2, 1);
HT_2.insert(cuco::pair{KEY_2, buf_idx_2});
BUF_2[buf_idx_2 * 1 + 0] = tid;
}
__global__ void count_5(uint64_t* COUNT4, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT4, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_INSERT HT_4, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
auto reg_date__d_datekey = date__d_datekey[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_4 = 0;

KEY_4 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_4 = atomicAdd((int*)BUF_IDX_4, 1);
HT_4.insert(cuco::pair{KEY_4, buf_idx_4});
BUF_4[buf_idx_4 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_7(uint64_t* BUF_0, uint64_t* BUF_2, uint64_t* BUF_4, HASHTABLE_PROBE HT_0, HASHTABLE_PROBE HT_2, HASHTABLE_PROBE HT_4, HASHTABLE_INSERT HT_6, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_0 = 0;

KEY_0 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_0.for_each(KEY_0, [&] __device__ (auto const SLOT_0) {

auto const [slot_first0, slot_second0] = SLOT_0;
if (!(true)) return;
uint64_t KEY_2 = 0;

KEY_2 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_2.for_each(KEY_2, [&] __device__ (auto const SLOT_2) {

auto const [slot_first2, slot_second2] = SLOT_2;
if (!(true)) return;
uint64_t KEY_4 = 0;

KEY_4 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_4.for_each(KEY_4, [&] __device__ (auto const SLOT_4) {

auto const [slot_first4, slot_second4] = SLOT_4;
if (!(true)) return;
uint64_t KEY_6 = 0;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[BUF_0[slot_second0 * 1 + 0]];

KEY_6 |= reg_customer__c_nation_encoded;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_2[slot_second2 * 1 + 0]];
KEY_6 <<= 16;
KEY_6 |= reg_supplier__s_nation_encoded;
auto reg_date__d_year = date__d_year[BUF_4[slot_second4 * 1 + 0]];
KEY_6 <<= 32;
KEY_6 |= reg_date__d_year;
//Create aggregation hash table
HT_6.insert(cuco::pair{KEY_6, 1});
});
});
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_7(uint64_t* BUF_0, uint64_t* BUF_2, uint64_t* BUF_4, HASHTABLE_PROBE HT_0, HASHTABLE_PROBE HT_2, HASHTABLE_PROBE HT_4, HASHTABLE_FIND HT_6, DBI16Type* KEY_6customer__c_nation_encoded, DBI32Type* KEY_6date__d_year, DBI16Type* KEY_6supplier__s_nation_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_0 = 0;

KEY_0 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_0.for_each(KEY_0, [&] __device__ (auto const SLOT_0) {
auto const [slot_first0, slot_second0] = SLOT_0;
if (!(true)) return;
uint64_t KEY_2 = 0;

KEY_2 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_2.for_each(KEY_2, [&] __device__ (auto const SLOT_2) {
auto const [slot_first2, slot_second2] = SLOT_2;
if (!(true)) return;
uint64_t KEY_4 = 0;

KEY_4 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_4.for_each(KEY_4, [&] __device__ (auto const SLOT_4) {
auto const [slot_first4, slot_second4] = SLOT_4;
if (!(true)) return;
uint64_t KEY_6 = 0;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[BUF_0[slot_second0 * 1 + 0]];

KEY_6 |= reg_customer__c_nation_encoded;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_2[slot_second2 * 1 + 0]];
KEY_6 <<= 16;
KEY_6 |= reg_supplier__s_nation_encoded;
auto reg_date__d_year = date__d_year[BUF_4[slot_second4 * 1 + 0]];
KEY_6 <<= 32;
KEY_6 |= reg_date__d_year;
//Aggregate in hashtable
auto buf_idx_6 = HT_6.find(KEY_6)->second;
aggregate_sum(&aggr0__tmp_attr0[buf_idx_6], reg_lineorder__lo_revenue);
KEY_6customer__c_nation_encoded[buf_idx_6] = reg_customer__c_nation_encoded;
KEY_6supplier__s_nation_encoded[buf_idx_6] = reg_supplier__s_nation_encoded;
KEY_6date__d_year[buf_idx_6] = reg_date__d_year;
});
});
});
}
__global__ void count_9(size_t COUNT6, uint64_t* COUNT8) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT6) return;
//Materialize count
atomicAdd((int*)COUNT8, 1);
}
__global__ void main_9(size_t COUNT6, DBDecimalType* MAT8aggr0__tmp_attr0, DBI16Type* MAT8customer__c_nation_encoded, DBI32Type* MAT8date__d_year, DBI16Type* MAT8supplier__s_nation_encoded, uint64_t* MAT_IDX8, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT6) return;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[tid];
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[tid];
auto reg_date__d_year = date__d_year[tid];
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
//Materialize buffers
auto mat_idx8 = atomicAdd((int*)MAT_IDX8, 1);
MAT8customer__c_nation_encoded[mat_idx8] = reg_customer__c_nation_encoded;
MAT8supplier__s_nation_encoded[mat_idx8] = reg_supplier__s_nation_encoded;
MAT8date__d_year[mat_idx8] = reg_date__d_year;
MAT8aggr0__tmp_attr0[mat_idx8] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)customer_size/128.), 128>>>(d_COUNT0, d_customer__c_region, customer_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::experimental::static_multimap{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_customer__c_custkey, d_customer__c_region, customer_size);
//Materialize count
uint64_t* d_COUNT2;
hipMalloc(&d_COUNT2, sizeof(uint64_t));
hipMemset(d_COUNT2, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT2, d_supplier__s_region, supplier_size);
uint64_t COUNT2;
hipMemcpy(&COUNT2, d_COUNT2, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 1);
auto d_HT_2 = cuco::experimental::static_multimap{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_2, d_BUF_IDX_2, d_HT_2.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT4;
hipMalloc(&d_COUNT4, sizeof(uint64_t));
hipMemset(d_COUNT4, 0, sizeof(uint64_t));
count_5<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT4, d_date__d_year, date_size);
uint64_t COUNT4;
hipMemcpy(&COUNT4, d_COUNT4, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 1);
auto d_HT_4 = cuco::experimental::static_multimap{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_4, d_BUF_IDX_4, d_HT_4.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Create aggregation hash table
auto d_HT_6 = cuco::static_map{ (int)149182*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_7<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_0, d_BUF_2, d_BUF_4, d_HT_0.ref(cuco::for_each), d_HT_2.ref(cuco::for_each), d_HT_4.ref(cuco::for_each), d_HT_6.ref(cuco::insert), d_customer__c_nation_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_nation_encoded);
size_t COUNT6 = d_HT_6.size();
thrust::device_vector<int64_t> keys_6(COUNT6), vals_6(COUNT6);
d_HT_6.retrieve_all(keys_6.begin(), vals_6.begin());
d_HT_6.clear();
int64_t* raw_keys6 = thrust::raw_pointer_cast(keys_6.data());
insertKeys<<<std::ceil((float)COUNT6/128.), 128>>>(raw_keys6, d_HT_6.ref(cuco::insert), COUNT6);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT6);
DBI16Type* d_KEY_6customer__c_nation_encoded;
hipMalloc(&d_KEY_6customer__c_nation_encoded, sizeof(DBI16Type) * COUNT6);
hipMemset(d_KEY_6customer__c_nation_encoded, 0, sizeof(DBI16Type) * COUNT6);
DBI16Type* d_KEY_6supplier__s_nation_encoded;
hipMalloc(&d_KEY_6supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT6);
hipMemset(d_KEY_6supplier__s_nation_encoded, 0, sizeof(DBI16Type) * COUNT6);
DBI32Type* d_KEY_6date__d_year;
hipMalloc(&d_KEY_6date__d_year, sizeof(DBI32Type) * COUNT6);
hipMemset(d_KEY_6date__d_year, 0, sizeof(DBI32Type) * COUNT6);
main_7<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_0, d_BUF_2, d_BUF_4, d_HT_0.ref(cuco::for_each), d_HT_2.ref(cuco::for_each), d_HT_4.ref(cuco::for_each), d_HT_6.ref(cuco::find), d_KEY_6customer__c_nation_encoded, d_KEY_6date__d_year, d_KEY_6supplier__s_nation_encoded, d_aggr0__tmp_attr0, d_customer__c_nation_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_nation_encoded);
//Materialize count
uint64_t* d_COUNT8;
hipMalloc(&d_COUNT8, sizeof(uint64_t));
hipMemset(d_COUNT8, 0, sizeof(uint64_t));
count_9<<<std::ceil((float)COUNT6/128.), 128>>>(COUNT6, d_COUNT8);
uint64_t COUNT8;
hipMemcpy(&COUNT8, d_COUNT8, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX8;
hipMalloc(&d_MAT_IDX8, sizeof(uint64_t));
hipMemset(d_MAT_IDX8, 0, sizeof(uint64_t));
auto MAT8customer__c_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT8);
DBI16Type* d_MAT8customer__c_nation_encoded;
hipMalloc(&d_MAT8customer__c_nation_encoded, sizeof(DBI16Type) * COUNT8);
auto MAT8supplier__s_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT8);
DBI16Type* d_MAT8supplier__s_nation_encoded;
hipMalloc(&d_MAT8supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT8);
auto MAT8date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT8);
DBI32Type* d_MAT8date__d_year;
hipMalloc(&d_MAT8date__d_year, sizeof(DBI32Type) * COUNT8);
auto MAT8aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT8);
DBDecimalType* d_MAT8aggr0__tmp_attr0;
hipMalloc(&d_MAT8aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT8);
main_9<<<std::ceil((float)COUNT6/128.), 128>>>(COUNT6, d_MAT8aggr0__tmp_attr0, d_MAT8customer__c_nation_encoded, d_MAT8date__d_year, d_MAT8supplier__s_nation_encoded, d_MAT_IDX8, d_aggr0__tmp_attr0, d_KEY_6customer__c_nation_encoded, d_KEY_6date__d_year, d_KEY_6supplier__s_nation_encoded);
hipMemcpy(MAT8customer__c_nation_encoded, d_MAT8customer__c_nation_encoded, sizeof(DBI16Type) * COUNT8, hipMemcpyDeviceToHost);
hipMemcpy(MAT8supplier__s_nation_encoded, d_MAT8supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT8, hipMemcpyDeviceToHost);
hipMemcpy(MAT8date__d_year, d_MAT8date__d_year, sizeof(DBI32Type) * COUNT8, hipMemcpyDeviceToHost);
hipMemcpy(MAT8aggr0__tmp_attr0, d_MAT8aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT8, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT8; i++) { std::cout << "" << customer__c_nation_map[MAT8customer__c_nation_encoded[i]];
std::cout << "," << supplier__s_nation_map[MAT8supplier__s_nation_encoded[i]];
std::cout << "," << MAT8date__d_year[i];
std::cout << "," << MAT8aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_COUNT0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_COUNT2);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_COUNT4);
hipFree(d_KEY_6customer__c_nation_encoded);
hipFree(d_KEY_6date__d_year);
hipFree(d_KEY_6supplier__s_nation_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT8);
hipFree(d_MAT8aggr0__tmp_attr0);
hipFree(d_MAT8customer__c_nation_encoded);
hipFree(d_MAT8date__d_year);
hipFree(d_MAT8supplier__s_nation_encoded);
hipFree(d_MAT_IDX8);
free(MAT8aggr0__tmp_attr0);
free(MAT8customer__c_nation_encoded);
free(MAT8date__d_year);
free(MAT8supplier__s_nation_encoded);
}