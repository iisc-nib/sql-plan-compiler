#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_640cd395fd00(uint64_t* COUNT640cd397dca0, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT640cd397dca0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_640cd395fd00(uint64_t* BUF_640cd397dca0, uint64_t* BUF_IDX_640cd397dca0, HASHTABLE_INSERT HT_640cd397dca0, DBI32Type* customer__c_custkey, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_640cd397dca0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_640cd397dca0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_640cd397dca0 = atomicAdd((int*)BUF_IDX_640cd397dca0, 1);
HT_640cd397dca0.insert(cuco::pair{KEY_640cd397dca0, buf_idx_640cd397dca0});
BUF_640cd397dca0[buf_idx_640cd397dca0 * 1 + 0] = tid;
}
__global__ void count_640cd3987040(uint64_t* COUNT640cd397c040, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT640cd397c040, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_640cd3987040(uint64_t* BUF_640cd397c040, uint64_t* BUF_IDX_640cd397c040, HASHTABLE_INSERT HT_640cd397c040, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_640cd397c040 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_640cd397c040 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_640cd397c040 = atomicAdd((int*)BUF_IDX_640cd397c040, 1);
HT_640cd397c040.insert(cuco::pair{KEY_640cd397c040, buf_idx_640cd397c040});
BUF_640cd397c040[buf_idx_640cd397c040 * 1 + 0] = tid;
}
__global__ void count_640cd398ceb0(uint64_t* COUNT640cd3980060, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT640cd3980060, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_640cd398ceb0(uint64_t* BUF_640cd3980060, uint64_t* BUF_IDX_640cd3980060, HASHTABLE_INSERT HT_640cd3980060, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_640cd3980060 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_640cd3980060 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_640cd3980060 = atomicAdd((int*)BUF_IDX_640cd3980060, 1);
HT_640cd3980060.insert(cuco::pair{KEY_640cd3980060, buf_idx_640cd3980060});
BUF_640cd3980060[buf_idx_640cd3980060 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_640cd39602a0(uint64_t* BUF_640cd397c040, uint64_t* BUF_640cd397dca0, uint64_t* BUF_640cd3980060, HASHTABLE_INSERT HT_640cd392f980, HASHTABLE_PROBE HT_640cd397c040, HASHTABLE_PROBE HT_640cd397dca0, HASHTABLE_PROBE HT_640cd3980060, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_640cd397dca0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_640cd397dca0 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_640cd397dca0.for_each(KEY_640cd397dca0, [&] __device__ (auto const SLOT_640cd397dca0) {

auto const [slot_first640cd397dca0, slot_second640cd397dca0] = SLOT_640cd397dca0;
if (!(true)) return;
uint64_t KEY_640cd397c040 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_640cd397c040 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_640cd397c040.for_each(KEY_640cd397c040, [&] __device__ (auto const SLOT_640cd397c040) {

auto const [slot_first640cd397c040, slot_second640cd397c040] = SLOT_640cd397c040;
if (!(true)) return;
uint64_t KEY_640cd3980060 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_640cd3980060 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_640cd3980060.for_each(KEY_640cd3980060, [&] __device__ (auto const SLOT_640cd3980060) {

auto const [slot_first640cd3980060, slot_second640cd3980060] = SLOT_640cd3980060;
if (!(true)) return;
uint64_t KEY_640cd392f980 = 0;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[BUF_640cd397dca0[slot_second640cd397dca0 * 1 + 0]];

KEY_640cd392f980 |= reg_customer__c_nation_encoded;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_640cd397c040[slot_second640cd397c040 * 1 + 0]];
KEY_640cd392f980 <<= 16;
KEY_640cd392f980 |= reg_supplier__s_nation_encoded;
auto reg_date__d_year = date__d_year[BUF_640cd3980060[slot_second640cd3980060 * 1 + 0]];
KEY_640cd392f980 <<= 32;
KEY_640cd392f980 |= reg_date__d_year;
//Create aggregation hash table
HT_640cd392f980.insert(cuco::pair{KEY_640cd392f980, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_640cd39602a0(uint64_t* BUF_640cd397c040, uint64_t* BUF_640cd397dca0, uint64_t* BUF_640cd3980060, HASHTABLE_FIND HT_640cd392f980, HASHTABLE_PROBE HT_640cd397c040, HASHTABLE_PROBE HT_640cd397dca0, HASHTABLE_PROBE HT_640cd3980060, DBI16Type* KEY_640cd392f980customer__c_nation_encoded, DBI32Type* KEY_640cd392f980date__d_year, DBI16Type* KEY_640cd392f980supplier__s_nation_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_640cd397dca0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_640cd397dca0 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_640cd397dca0.for_each(KEY_640cd397dca0, [&] __device__ (auto const SLOT_640cd397dca0) {
auto const [slot_first640cd397dca0, slot_second640cd397dca0] = SLOT_640cd397dca0;
if (!(true)) return;
uint64_t KEY_640cd397c040 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_640cd397c040 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_640cd397c040.for_each(KEY_640cd397c040, [&] __device__ (auto const SLOT_640cd397c040) {
auto const [slot_first640cd397c040, slot_second640cd397c040] = SLOT_640cd397c040;
if (!(true)) return;
uint64_t KEY_640cd3980060 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_640cd3980060 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_640cd3980060.for_each(KEY_640cd3980060, [&] __device__ (auto const SLOT_640cd3980060) {
auto const [slot_first640cd3980060, slot_second640cd3980060] = SLOT_640cd3980060;
if (!(true)) return;
uint64_t KEY_640cd392f980 = 0;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[BUF_640cd397dca0[slot_second640cd397dca0 * 1 + 0]];

KEY_640cd392f980 |= reg_customer__c_nation_encoded;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_640cd397c040[slot_second640cd397c040 * 1 + 0]];
KEY_640cd392f980 <<= 16;
KEY_640cd392f980 |= reg_supplier__s_nation_encoded;
auto reg_date__d_year = date__d_year[BUF_640cd3980060[slot_second640cd3980060 * 1 + 0]];
KEY_640cd392f980 <<= 32;
KEY_640cd392f980 |= reg_date__d_year;
//Aggregate in hashtable
auto buf_idx_640cd392f980 = HT_640cd392f980.find(KEY_640cd392f980)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_640cd392f980], reg_lineorder__lo_revenue);
KEY_640cd392f980customer__c_nation_encoded[buf_idx_640cd392f980] = reg_customer__c_nation_encoded;
KEY_640cd392f980supplier__s_nation_encoded[buf_idx_640cd392f980] = reg_supplier__s_nation_encoded;
KEY_640cd392f980date__d_year[buf_idx_640cd392f980] = reg_date__d_year;
});
});
});
}
__global__ void count_640cd399a370(uint64_t* COUNT640cd390b830, size_t COUNT640cd392f980) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT640cd392f980) return;
//Materialize count
atomicAdd((int*)COUNT640cd390b830, 1);
}
__global__ void main_640cd399a370(size_t COUNT640cd392f980, DBDecimalType* MAT640cd390b830aggr0__tmp_attr0, DBI16Type* MAT640cd390b830customer__c_nation_encoded, DBI32Type* MAT640cd390b830date__d_year, DBI16Type* MAT640cd390b830supplier__s_nation_encoded, uint64_t* MAT_IDX640cd390b830, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT640cd392f980) return;
//Materialize buffers
auto mat_idx640cd390b830 = atomicAdd((int*)MAT_IDX640cd390b830, 1);
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[tid];
MAT640cd390b830customer__c_nation_encoded[mat_idx640cd390b830] = reg_customer__c_nation_encoded;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[tid];
MAT640cd390b830supplier__s_nation_encoded[mat_idx640cd390b830] = reg_supplier__s_nation_encoded;
auto reg_date__d_year = date__d_year[tid];
MAT640cd390b830date__d_year[mat_idx640cd390b830] = reg_date__d_year;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT640cd390b830aggr0__tmp_attr0[mat_idx640cd390b830] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT640cd397dca0;
hipMalloc(&d_COUNT640cd397dca0, sizeof(uint64_t));
hipMemset(d_COUNT640cd397dca0, 0, sizeof(uint64_t));
count_640cd395fd00<<<std::ceil((float)customer_size/128.), 128>>>(d_COUNT640cd397dca0, d_customer__c_region, customer_size);
uint64_t COUNT640cd397dca0;
hipMemcpy(&COUNT640cd397dca0, d_COUNT640cd397dca0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_640cd397dca0;
hipMalloc(&d_BUF_IDX_640cd397dca0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_640cd397dca0, 0, sizeof(uint64_t));
uint64_t* d_BUF_640cd397dca0;
hipMalloc(&d_BUF_640cd397dca0, sizeof(uint64_t) * COUNT640cd397dca0 * 1);
auto d_HT_640cd397dca0 = cuco::experimental::static_multimap{ (int)COUNT640cd397dca0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_640cd395fd00<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_640cd397dca0, d_BUF_IDX_640cd397dca0, d_HT_640cd397dca0.ref(cuco::insert), d_customer__c_custkey, d_customer__c_region, customer_size);
//Materialize count
uint64_t* d_COUNT640cd397c040;
hipMalloc(&d_COUNT640cd397c040, sizeof(uint64_t));
hipMemset(d_COUNT640cd397c040, 0, sizeof(uint64_t));
count_640cd3987040<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT640cd397c040, d_supplier__s_region, supplier_size);
uint64_t COUNT640cd397c040;
hipMemcpy(&COUNT640cd397c040, d_COUNT640cd397c040, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_640cd397c040;
hipMalloc(&d_BUF_IDX_640cd397c040, sizeof(uint64_t));
hipMemset(d_BUF_IDX_640cd397c040, 0, sizeof(uint64_t));
uint64_t* d_BUF_640cd397c040;
hipMalloc(&d_BUF_640cd397c040, sizeof(uint64_t) * COUNT640cd397c040 * 1);
auto d_HT_640cd397c040 = cuco::experimental::static_multimap{ (int)COUNT640cd397c040*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_640cd3987040<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_640cd397c040, d_BUF_IDX_640cd397c040, d_HT_640cd397c040.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT640cd3980060;
hipMalloc(&d_COUNT640cd3980060, sizeof(uint64_t));
hipMemset(d_COUNT640cd3980060, 0, sizeof(uint64_t));
count_640cd398ceb0<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT640cd3980060, d_date__d_year, date_size);
uint64_t COUNT640cd3980060;
hipMemcpy(&COUNT640cd3980060, d_COUNT640cd3980060, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_640cd3980060;
hipMalloc(&d_BUF_IDX_640cd3980060, sizeof(uint64_t));
hipMemset(d_BUF_IDX_640cd3980060, 0, sizeof(uint64_t));
uint64_t* d_BUF_640cd3980060;
hipMalloc(&d_BUF_640cd3980060, sizeof(uint64_t) * COUNT640cd3980060 * 1);
auto d_HT_640cd3980060 = cuco::experimental::static_multimap{ (int)COUNT640cd3980060*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_640cd398ceb0<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_640cd3980060, d_BUF_IDX_640cd3980060, d_HT_640cd3980060.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Create aggregation hash table
auto d_HT_640cd392f980 = cuco::static_map{ (int)144285*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_640cd39602a0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_640cd397c040, d_BUF_640cd397dca0, d_BUF_640cd3980060, d_HT_640cd392f980.ref(cuco::insert), d_HT_640cd397c040.ref(cuco::for_each), d_HT_640cd397dca0.ref(cuco::for_each), d_HT_640cd3980060.ref(cuco::for_each), d_customer__c_nation_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_nation_encoded);
size_t COUNT640cd392f980 = d_HT_640cd392f980.size();
thrust::device_vector<int64_t> keys_640cd392f980(COUNT640cd392f980), vals_640cd392f980(COUNT640cd392f980);
d_HT_640cd392f980.retrieve_all(keys_640cd392f980.begin(), vals_640cd392f980.begin());
d_HT_640cd392f980.clear();
int64_t* raw_keys640cd392f980 = thrust::raw_pointer_cast(keys_640cd392f980.data());
insertKeys<<<std::ceil((float)COUNT640cd392f980/128.), 128>>>(raw_keys640cd392f980, d_HT_640cd392f980.ref(cuco::insert), COUNT640cd392f980);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT640cd392f980);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT640cd392f980);
DBI16Type* d_KEY_640cd392f980customer__c_nation_encoded;
hipMalloc(&d_KEY_640cd392f980customer__c_nation_encoded, sizeof(DBI16Type) * COUNT640cd392f980);
hipMemset(d_KEY_640cd392f980customer__c_nation_encoded, 0, sizeof(DBI16Type) * COUNT640cd392f980);
DBI16Type* d_KEY_640cd392f980supplier__s_nation_encoded;
hipMalloc(&d_KEY_640cd392f980supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT640cd392f980);
hipMemset(d_KEY_640cd392f980supplier__s_nation_encoded, 0, sizeof(DBI16Type) * COUNT640cd392f980);
DBI32Type* d_KEY_640cd392f980date__d_year;
hipMalloc(&d_KEY_640cd392f980date__d_year, sizeof(DBI32Type) * COUNT640cd392f980);
hipMemset(d_KEY_640cd392f980date__d_year, 0, sizeof(DBI32Type) * COUNT640cd392f980);
main_640cd39602a0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_640cd397c040, d_BUF_640cd397dca0, d_BUF_640cd3980060, d_HT_640cd392f980.ref(cuco::find), d_HT_640cd397c040.ref(cuco::for_each), d_HT_640cd397dca0.ref(cuco::for_each), d_HT_640cd3980060.ref(cuco::for_each), d_KEY_640cd392f980customer__c_nation_encoded, d_KEY_640cd392f980date__d_year, d_KEY_640cd392f980supplier__s_nation_encoded, d_aggr0__tmp_attr0, d_customer__c_nation_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_nation_encoded);
//Materialize count
uint64_t* d_COUNT640cd390b830;
hipMalloc(&d_COUNT640cd390b830, sizeof(uint64_t));
hipMemset(d_COUNT640cd390b830, 0, sizeof(uint64_t));
count_640cd399a370<<<std::ceil((float)COUNT640cd392f980/128.), 128>>>(d_COUNT640cd390b830, COUNT640cd392f980);
uint64_t COUNT640cd390b830;
hipMemcpy(&COUNT640cd390b830, d_COUNT640cd390b830, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX640cd390b830;
hipMalloc(&d_MAT_IDX640cd390b830, sizeof(uint64_t));
hipMemset(d_MAT_IDX640cd390b830, 0, sizeof(uint64_t));
auto MAT640cd390b830customer__c_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT640cd390b830);
DBI16Type* d_MAT640cd390b830customer__c_nation_encoded;
hipMalloc(&d_MAT640cd390b830customer__c_nation_encoded, sizeof(DBI16Type) * COUNT640cd390b830);
auto MAT640cd390b830supplier__s_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT640cd390b830);
DBI16Type* d_MAT640cd390b830supplier__s_nation_encoded;
hipMalloc(&d_MAT640cd390b830supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT640cd390b830);
auto MAT640cd390b830date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT640cd390b830);
DBI32Type* d_MAT640cd390b830date__d_year;
hipMalloc(&d_MAT640cd390b830date__d_year, sizeof(DBI32Type) * COUNT640cd390b830);
auto MAT640cd390b830aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT640cd390b830);
DBDecimalType* d_MAT640cd390b830aggr0__tmp_attr0;
hipMalloc(&d_MAT640cd390b830aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT640cd390b830);
main_640cd399a370<<<std::ceil((float)COUNT640cd392f980/128.), 128>>>(COUNT640cd392f980, d_MAT640cd390b830aggr0__tmp_attr0, d_MAT640cd390b830customer__c_nation_encoded, d_MAT640cd390b830date__d_year, d_MAT640cd390b830supplier__s_nation_encoded, d_MAT_IDX640cd390b830, d_aggr0__tmp_attr0, d_KEY_640cd392f980customer__c_nation_encoded, d_KEY_640cd392f980date__d_year, d_KEY_640cd392f980supplier__s_nation_encoded);
hipMemcpy(MAT640cd390b830customer__c_nation_encoded, d_MAT640cd390b830customer__c_nation_encoded, sizeof(DBI16Type) * COUNT640cd390b830, hipMemcpyDeviceToHost);
hipMemcpy(MAT640cd390b830supplier__s_nation_encoded, d_MAT640cd390b830supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT640cd390b830, hipMemcpyDeviceToHost);
hipMemcpy(MAT640cd390b830date__d_year, d_MAT640cd390b830date__d_year, sizeof(DBI32Type) * COUNT640cd390b830, hipMemcpyDeviceToHost);
hipMemcpy(MAT640cd390b830aggr0__tmp_attr0, d_MAT640cd390b830aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT640cd390b830, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT640cd390b830; i++) { std::cout << "" << customer__c_nation_map[MAT640cd390b830customer__c_nation_encoded[i]];
std::cout << "," << supplier__s_nation_map[MAT640cd390b830supplier__s_nation_encoded[i]];
std::cout << "," << MAT640cd390b830date__d_year[i];
std::cout << "," << MAT640cd390b830aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_640cd397dca0);
hipFree(d_BUF_IDX_640cd397dca0);
hipFree(d_COUNT640cd397dca0);
hipFree(d_BUF_640cd397c040);
hipFree(d_BUF_IDX_640cd397c040);
hipFree(d_COUNT640cd397c040);
hipFree(d_BUF_640cd3980060);
hipFree(d_BUF_IDX_640cd3980060);
hipFree(d_COUNT640cd3980060);
hipFree(d_KEY_640cd392f980customer__c_nation_encoded);
hipFree(d_KEY_640cd392f980date__d_year);
hipFree(d_KEY_640cd392f980supplier__s_nation_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT640cd390b830);
hipFree(d_MAT640cd390b830aggr0__tmp_attr0);
hipFree(d_MAT640cd390b830customer__c_nation_encoded);
hipFree(d_MAT640cd390b830date__d_year);
hipFree(d_MAT640cd390b830supplier__s_nation_encoded);
hipFree(d_MAT_IDX640cd390b830);
free(MAT640cd390b830aggr0__tmp_attr0);
free(MAT640cd390b830customer__c_nation_encoded);
free(MAT640cd390b830date__d_year);
free(MAT640cd390b830supplier__s_nation_encoded);
}