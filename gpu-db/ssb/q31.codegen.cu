#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5cc9c1baede0(uint64_t* COUNT5cc9c1bca9e0, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5cc9c1bca9e0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5cc9c1baede0(uint64_t* BUF_5cc9c1bca9e0, uint64_t* BUF_IDX_5cc9c1bca9e0, HASHTABLE_INSERT HT_5cc9c1bca9e0, DBI32Type* customer__c_custkey, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5cc9c1bca9e0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5cc9c1bca9e0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_5cc9c1bca9e0 = atomicAdd((int*)BUF_IDX_5cc9c1bca9e0, 1);
HT_5cc9c1bca9e0.insert(cuco::pair{KEY_5cc9c1bca9e0, buf_idx_5cc9c1bca9e0});
BUF_5cc9c1bca9e0[buf_idx_5cc9c1bca9e0 * 1 + 0] = tid;
}
__global__ void count_5cc9c1bd57c0(uint64_t* COUNT5cc9c1bccc30, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5cc9c1bccc30, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5cc9c1bd57c0(uint64_t* BUF_5cc9c1bccc30, uint64_t* BUF_IDX_5cc9c1bccc30, HASHTABLE_INSERT HT_5cc9c1bccc30, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5cc9c1bccc30 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5cc9c1bccc30 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5cc9c1bccc30 = atomicAdd((int*)BUF_IDX_5cc9c1bccc30, 1);
HT_5cc9c1bccc30.insert(cuco::pair{KEY_5cc9c1bccc30, buf_idx_5cc9c1bccc30});
BUF_5cc9c1bccc30[buf_idx_5cc9c1bccc30 * 1 + 0] = tid;
}
__global__ void count_5cc9c1bdb120(uint64_t* COUNT5cc9c1bca380, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5cc9c1bca380, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5cc9c1bdb120(uint64_t* BUF_5cc9c1bca380, uint64_t* BUF_IDX_5cc9c1bca380, HASHTABLE_INSERT HT_5cc9c1bca380, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5cc9c1bca380 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5cc9c1bca380 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_5cc9c1bca380 = atomicAdd((int*)BUF_IDX_5cc9c1bca380, 1);
HT_5cc9c1bca380.insert(cuco::pair{KEY_5cc9c1bca380, buf_idx_5cc9c1bca380});
BUF_5cc9c1bca380[buf_idx_5cc9c1bca380 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5cc9c1baf380(uint64_t* BUF_5cc9c1bca380, uint64_t* BUF_5cc9c1bca9e0, uint64_t* BUF_5cc9c1bccc30, HASHTABLE_INSERT HT_5cc9c1b7e660, HASHTABLE_PROBE HT_5cc9c1bca380, HASHTABLE_PROBE HT_5cc9c1bca9e0, HASHTABLE_PROBE HT_5cc9c1bccc30, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5cc9c1bca9e0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5cc9c1bca9e0 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_5cc9c1bca9e0.for_each(KEY_5cc9c1bca9e0, [&] __device__ (auto const SLOT_5cc9c1bca9e0) {

auto const [slot_first5cc9c1bca9e0, slot_second5cc9c1bca9e0] = SLOT_5cc9c1bca9e0;
if (!(true)) return;
uint64_t KEY_5cc9c1bccc30 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5cc9c1bccc30 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5cc9c1bccc30.for_each(KEY_5cc9c1bccc30, [&] __device__ (auto const SLOT_5cc9c1bccc30) {

auto const [slot_first5cc9c1bccc30, slot_second5cc9c1bccc30] = SLOT_5cc9c1bccc30;
if (!(true)) return;
uint64_t KEY_5cc9c1bca380 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5cc9c1bca380 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5cc9c1bca380.for_each(KEY_5cc9c1bca380, [&] __device__ (auto const SLOT_5cc9c1bca380) {

auto const [slot_first5cc9c1bca380, slot_second5cc9c1bca380] = SLOT_5cc9c1bca380;
if (!(true)) return;
uint64_t KEY_5cc9c1b7e660 = 0;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[BUF_5cc9c1bca9e0[slot_second5cc9c1bca9e0 * 1 + 0]];

KEY_5cc9c1b7e660 |= reg_customer__c_nation_encoded;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_5cc9c1bccc30[slot_second5cc9c1bccc30 * 1 + 0]];
KEY_5cc9c1b7e660 <<= 16;
KEY_5cc9c1b7e660 |= reg_supplier__s_nation_encoded;
auto reg_date__d_year = date__d_year[BUF_5cc9c1bca380[slot_second5cc9c1bca380 * 1 + 0]];
KEY_5cc9c1b7e660 <<= 32;
KEY_5cc9c1b7e660 |= reg_date__d_year;
//Create aggregation hash table
HT_5cc9c1b7e660.insert(cuco::pair{KEY_5cc9c1b7e660, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5cc9c1baf380(uint64_t* BUF_5cc9c1bca380, uint64_t* BUF_5cc9c1bca9e0, uint64_t* BUF_5cc9c1bccc30, HASHTABLE_FIND HT_5cc9c1b7e660, HASHTABLE_PROBE HT_5cc9c1bca380, HASHTABLE_PROBE HT_5cc9c1bca9e0, HASHTABLE_PROBE HT_5cc9c1bccc30, DBI16Type* KEY_5cc9c1b7e660customer__c_nation_encoded, DBI32Type* KEY_5cc9c1b7e660date__d_year, DBI16Type* KEY_5cc9c1b7e660supplier__s_nation_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5cc9c1bca9e0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5cc9c1bca9e0 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_5cc9c1bca9e0.for_each(KEY_5cc9c1bca9e0, [&] __device__ (auto const SLOT_5cc9c1bca9e0) {
auto const [slot_first5cc9c1bca9e0, slot_second5cc9c1bca9e0] = SLOT_5cc9c1bca9e0;
if (!(true)) return;
uint64_t KEY_5cc9c1bccc30 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5cc9c1bccc30 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5cc9c1bccc30.for_each(KEY_5cc9c1bccc30, [&] __device__ (auto const SLOT_5cc9c1bccc30) {
auto const [slot_first5cc9c1bccc30, slot_second5cc9c1bccc30] = SLOT_5cc9c1bccc30;
if (!(true)) return;
uint64_t KEY_5cc9c1bca380 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5cc9c1bca380 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5cc9c1bca380.for_each(KEY_5cc9c1bca380, [&] __device__ (auto const SLOT_5cc9c1bca380) {
auto const [slot_first5cc9c1bca380, slot_second5cc9c1bca380] = SLOT_5cc9c1bca380;
if (!(true)) return;
uint64_t KEY_5cc9c1b7e660 = 0;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[BUF_5cc9c1bca9e0[slot_second5cc9c1bca9e0 * 1 + 0]];

KEY_5cc9c1b7e660 |= reg_customer__c_nation_encoded;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_5cc9c1bccc30[slot_second5cc9c1bccc30 * 1 + 0]];
KEY_5cc9c1b7e660 <<= 16;
KEY_5cc9c1b7e660 |= reg_supplier__s_nation_encoded;
auto reg_date__d_year = date__d_year[BUF_5cc9c1bca380[slot_second5cc9c1bca380 * 1 + 0]];
KEY_5cc9c1b7e660 <<= 32;
KEY_5cc9c1b7e660 |= reg_date__d_year;
//Aggregate in hashtable
auto buf_idx_5cc9c1b7e660 = HT_5cc9c1b7e660.find(KEY_5cc9c1b7e660)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5cc9c1b7e660], reg_lineorder__lo_revenue);
KEY_5cc9c1b7e660customer__c_nation_encoded[buf_idx_5cc9c1b7e660] = reg_customer__c_nation_encoded;
KEY_5cc9c1b7e660supplier__s_nation_encoded[buf_idx_5cc9c1b7e660] = reg_supplier__s_nation_encoded;
KEY_5cc9c1b7e660date__d_year[buf_idx_5cc9c1b7e660] = reg_date__d_year;
});
});
});
}
__global__ void count_5cc9c1be82a0(uint64_t* COUNT5cc9c1b5a020, size_t COUNT5cc9c1b7e660) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5cc9c1b7e660) return;
//Materialize count
atomicAdd((int*)COUNT5cc9c1b5a020, 1);
}
__global__ void main_5cc9c1be82a0(size_t COUNT5cc9c1b7e660, DBDecimalType* MAT5cc9c1b5a020aggr0__tmp_attr0, DBI16Type* MAT5cc9c1b5a020customer__c_nation_encoded, DBI32Type* MAT5cc9c1b5a020date__d_year, DBI16Type* MAT5cc9c1b5a020supplier__s_nation_encoded, uint64_t* MAT_IDX5cc9c1b5a020, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5cc9c1b7e660) return;
//Materialize buffers
auto mat_idx5cc9c1b5a020 = atomicAdd((int*)MAT_IDX5cc9c1b5a020, 1);
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[tid];
MAT5cc9c1b5a020customer__c_nation_encoded[mat_idx5cc9c1b5a020] = reg_customer__c_nation_encoded;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[tid];
MAT5cc9c1b5a020supplier__s_nation_encoded[mat_idx5cc9c1b5a020] = reg_supplier__s_nation_encoded;
auto reg_date__d_year = date__d_year[tid];
MAT5cc9c1b5a020date__d_year[mat_idx5cc9c1b5a020] = reg_date__d_year;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5cc9c1b5a020aggr0__tmp_attr0[mat_idx5cc9c1b5a020] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map) {
//Materialize count
uint64_t* d_COUNT5cc9c1bca9e0;
hipMalloc(&d_COUNT5cc9c1bca9e0, sizeof(uint64_t));
hipMemset(d_COUNT5cc9c1bca9e0, 0, sizeof(uint64_t));
count_5cc9c1baede0<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT5cc9c1bca9e0, d_customer__c_region, customer_size);
uint64_t COUNT5cc9c1bca9e0;
hipMemcpy(&COUNT5cc9c1bca9e0, d_COUNT5cc9c1bca9e0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5cc9c1bca9e0;
hipMalloc(&d_BUF_IDX_5cc9c1bca9e0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5cc9c1bca9e0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5cc9c1bca9e0;
hipMalloc(&d_BUF_5cc9c1bca9e0, sizeof(uint64_t) * COUNT5cc9c1bca9e0 * 1);
auto d_HT_5cc9c1bca9e0 = cuco::experimental::static_multimap{ (int)COUNT5cc9c1bca9e0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5cc9c1baede0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5cc9c1bca9e0, d_BUF_IDX_5cc9c1bca9e0, d_HT_5cc9c1bca9e0.ref(cuco::insert), d_customer__c_custkey, d_customer__c_region, customer_size);
//Materialize count
uint64_t* d_COUNT5cc9c1bccc30;
hipMalloc(&d_COUNT5cc9c1bccc30, sizeof(uint64_t));
hipMemset(d_COUNT5cc9c1bccc30, 0, sizeof(uint64_t));
count_5cc9c1bd57c0<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT5cc9c1bccc30, d_supplier__s_region, supplier_size);
uint64_t COUNT5cc9c1bccc30;
hipMemcpy(&COUNT5cc9c1bccc30, d_COUNT5cc9c1bccc30, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5cc9c1bccc30;
hipMalloc(&d_BUF_IDX_5cc9c1bccc30, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5cc9c1bccc30, 0, sizeof(uint64_t));
uint64_t* d_BUF_5cc9c1bccc30;
hipMalloc(&d_BUF_5cc9c1bccc30, sizeof(uint64_t) * COUNT5cc9c1bccc30 * 1);
auto d_HT_5cc9c1bccc30 = cuco::experimental::static_multimap{ (int)COUNT5cc9c1bccc30*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5cc9c1bd57c0<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_5cc9c1bccc30, d_BUF_IDX_5cc9c1bccc30, d_HT_5cc9c1bccc30.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5cc9c1bca380;
hipMalloc(&d_COUNT5cc9c1bca380, sizeof(uint64_t));
hipMemset(d_COUNT5cc9c1bca380, 0, sizeof(uint64_t));
count_5cc9c1bdb120<<<std::ceil((float)date_size/32.), 32>>>(d_COUNT5cc9c1bca380, d_date__d_year, date_size);
uint64_t COUNT5cc9c1bca380;
hipMemcpy(&COUNT5cc9c1bca380, d_COUNT5cc9c1bca380, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5cc9c1bca380;
hipMalloc(&d_BUF_IDX_5cc9c1bca380, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5cc9c1bca380, 0, sizeof(uint64_t));
uint64_t* d_BUF_5cc9c1bca380;
hipMalloc(&d_BUF_5cc9c1bca380, sizeof(uint64_t) * COUNT5cc9c1bca380 * 1);
auto d_HT_5cc9c1bca380 = cuco::experimental::static_multimap{ (int)COUNT5cc9c1bca380*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5cc9c1bdb120<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_5cc9c1bca380, d_BUF_IDX_5cc9c1bca380, d_HT_5cc9c1bca380.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Create aggregation hash table
auto d_HT_5cc9c1b7e660 = cuco::static_map{ (int)144285*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5cc9c1baf380<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_5cc9c1bca380, d_BUF_5cc9c1bca9e0, d_BUF_5cc9c1bccc30, d_HT_5cc9c1b7e660.ref(cuco::insert), d_HT_5cc9c1bca380.ref(cuco::for_each), d_HT_5cc9c1bca9e0.ref(cuco::for_each), d_HT_5cc9c1bccc30.ref(cuco::for_each), d_customer__c_nation_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_nation_encoded);
size_t COUNT5cc9c1b7e660 = d_HT_5cc9c1b7e660.size();
thrust::device_vector<int64_t> keys_5cc9c1b7e660(COUNT5cc9c1b7e660), vals_5cc9c1b7e660(COUNT5cc9c1b7e660);
d_HT_5cc9c1b7e660.retrieve_all(keys_5cc9c1b7e660.begin(), vals_5cc9c1b7e660.begin());
d_HT_5cc9c1b7e660.clear();
int64_t* raw_keys5cc9c1b7e660 = thrust::raw_pointer_cast(keys_5cc9c1b7e660.data());
insertKeys<<<std::ceil((float)COUNT5cc9c1b7e660/32.), 32>>>(raw_keys5cc9c1b7e660, d_HT_5cc9c1b7e660.ref(cuco::insert), COUNT5cc9c1b7e660);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5cc9c1b7e660);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5cc9c1b7e660);
DBI16Type* d_KEY_5cc9c1b7e660customer__c_nation_encoded;
hipMalloc(&d_KEY_5cc9c1b7e660customer__c_nation_encoded, sizeof(DBI16Type) * COUNT5cc9c1b7e660);
hipMemset(d_KEY_5cc9c1b7e660customer__c_nation_encoded, 0, sizeof(DBI16Type) * COUNT5cc9c1b7e660);
DBI16Type* d_KEY_5cc9c1b7e660supplier__s_nation_encoded;
hipMalloc(&d_KEY_5cc9c1b7e660supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT5cc9c1b7e660);
hipMemset(d_KEY_5cc9c1b7e660supplier__s_nation_encoded, 0, sizeof(DBI16Type) * COUNT5cc9c1b7e660);
DBI32Type* d_KEY_5cc9c1b7e660date__d_year;
hipMalloc(&d_KEY_5cc9c1b7e660date__d_year, sizeof(DBI32Type) * COUNT5cc9c1b7e660);
hipMemset(d_KEY_5cc9c1b7e660date__d_year, 0, sizeof(DBI32Type) * COUNT5cc9c1b7e660);
main_5cc9c1baf380<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_5cc9c1bca380, d_BUF_5cc9c1bca9e0, d_BUF_5cc9c1bccc30, d_HT_5cc9c1b7e660.ref(cuco::find), d_HT_5cc9c1bca380.ref(cuco::for_each), d_HT_5cc9c1bca9e0.ref(cuco::for_each), d_HT_5cc9c1bccc30.ref(cuco::for_each), d_KEY_5cc9c1b7e660customer__c_nation_encoded, d_KEY_5cc9c1b7e660date__d_year, d_KEY_5cc9c1b7e660supplier__s_nation_encoded, d_aggr0__tmp_attr0, d_customer__c_nation_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_nation_encoded);
//Materialize count
uint64_t* d_COUNT5cc9c1b5a020;
hipMalloc(&d_COUNT5cc9c1b5a020, sizeof(uint64_t));
hipMemset(d_COUNT5cc9c1b5a020, 0, sizeof(uint64_t));
count_5cc9c1be82a0<<<std::ceil((float)COUNT5cc9c1b7e660/32.), 32>>>(d_COUNT5cc9c1b5a020, COUNT5cc9c1b7e660);
uint64_t COUNT5cc9c1b5a020;
hipMemcpy(&COUNT5cc9c1b5a020, d_COUNT5cc9c1b5a020, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5cc9c1b5a020;
hipMalloc(&d_MAT_IDX5cc9c1b5a020, sizeof(uint64_t));
hipMemset(d_MAT_IDX5cc9c1b5a020, 0, sizeof(uint64_t));
auto MAT5cc9c1b5a020customer__c_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5cc9c1b5a020);
DBI16Type* d_MAT5cc9c1b5a020customer__c_nation_encoded;
hipMalloc(&d_MAT5cc9c1b5a020customer__c_nation_encoded, sizeof(DBI16Type) * COUNT5cc9c1b5a020);
auto MAT5cc9c1b5a020supplier__s_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5cc9c1b5a020);
DBI16Type* d_MAT5cc9c1b5a020supplier__s_nation_encoded;
hipMalloc(&d_MAT5cc9c1b5a020supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT5cc9c1b5a020);
auto MAT5cc9c1b5a020date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5cc9c1b5a020);
DBI32Type* d_MAT5cc9c1b5a020date__d_year;
hipMalloc(&d_MAT5cc9c1b5a020date__d_year, sizeof(DBI32Type) * COUNT5cc9c1b5a020);
auto MAT5cc9c1b5a020aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5cc9c1b5a020);
DBDecimalType* d_MAT5cc9c1b5a020aggr0__tmp_attr0;
hipMalloc(&d_MAT5cc9c1b5a020aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5cc9c1b5a020);
main_5cc9c1be82a0<<<std::ceil((float)COUNT5cc9c1b7e660/32.), 32>>>(COUNT5cc9c1b7e660, d_MAT5cc9c1b5a020aggr0__tmp_attr0, d_MAT5cc9c1b5a020customer__c_nation_encoded, d_MAT5cc9c1b5a020date__d_year, d_MAT5cc9c1b5a020supplier__s_nation_encoded, d_MAT_IDX5cc9c1b5a020, d_aggr0__tmp_attr0, d_KEY_5cc9c1b7e660customer__c_nation_encoded, d_KEY_5cc9c1b7e660date__d_year, d_KEY_5cc9c1b7e660supplier__s_nation_encoded);
hipMemcpy(MAT5cc9c1b5a020customer__c_nation_encoded, d_MAT5cc9c1b5a020customer__c_nation_encoded, sizeof(DBI16Type) * COUNT5cc9c1b5a020, hipMemcpyDeviceToHost);
hipMemcpy(MAT5cc9c1b5a020supplier__s_nation_encoded, d_MAT5cc9c1b5a020supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT5cc9c1b5a020, hipMemcpyDeviceToHost);
hipMemcpy(MAT5cc9c1b5a020date__d_year, d_MAT5cc9c1b5a020date__d_year, sizeof(DBI32Type) * COUNT5cc9c1b5a020, hipMemcpyDeviceToHost);
hipMemcpy(MAT5cc9c1b5a020aggr0__tmp_attr0, d_MAT5cc9c1b5a020aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5cc9c1b5a020, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5cc9c1b5a020; i++) { std::cout << customer__c_nation_map[MAT5cc9c1b5a020customer__c_nation_encoded[i]] << "\t";
std::cout << supplier__s_nation_map[MAT5cc9c1b5a020supplier__s_nation_encoded[i]] << "\t";
std::cout << MAT5cc9c1b5a020date__d_year[i] << "\t";
std::cout << MAT5cc9c1b5a020aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_5cc9c1bca9e0);
hipFree(d_BUF_IDX_5cc9c1bca9e0);
hipFree(d_COUNT5cc9c1bca9e0);
hipFree(d_BUF_5cc9c1bccc30);
hipFree(d_BUF_IDX_5cc9c1bccc30);
hipFree(d_COUNT5cc9c1bccc30);
hipFree(d_BUF_5cc9c1bca380);
hipFree(d_BUF_IDX_5cc9c1bca380);
hipFree(d_COUNT5cc9c1bca380);
hipFree(d_KEY_5cc9c1b7e660customer__c_nation_encoded);
hipFree(d_KEY_5cc9c1b7e660date__d_year);
hipFree(d_KEY_5cc9c1b7e660supplier__s_nation_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5cc9c1b5a020);
hipFree(d_MAT5cc9c1b5a020aggr0__tmp_attr0);
hipFree(d_MAT5cc9c1b5a020customer__c_nation_encoded);
hipFree(d_MAT5cc9c1b5a020date__d_year);
hipFree(d_MAT5cc9c1b5a020supplier__s_nation_encoded);
hipFree(d_MAT_IDX5cc9c1b5a020);
free(MAT5cc9c1b5a020aggr0__tmp_attr0);
free(MAT5cc9c1b5a020customer__c_nation_encoded);
free(MAT5cc9c1b5a020date__d_year);
free(MAT5cc9c1b5a020supplier__s_nation_encoded);
}