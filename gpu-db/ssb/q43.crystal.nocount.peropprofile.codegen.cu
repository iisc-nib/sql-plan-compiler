#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_INSERT>
__global__ void main_8(uint64_t* BUF_28, uint64_t* BUF_IDX_28, HASHTABLE_INSERT HT_28, int64_t* cycles_per_warp_main_8_join_build_28, int64_t* cycles_per_warp_main_8_selection_10, int64_t* cycles_per_warp_main_8_selection_11, int64_t* cycles_per_warp_main_8_selection_12, int64_t* cycles_per_warp_main_8_selection_7, int64_t* cycles_per_warp_main_8_selection_9, DBStringType* supplier__s_nation, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_supplier__s_nation[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_nation[ITEM] = supplier__s_nation[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_supplier__s_nation[ITEM], "UNITED STATES", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_8_selection_7[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_8_selection_9[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_8_selection_10[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_8_selection_11[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_8_selection_12[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_28[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_28[ITEM] = 0;
KEY_28[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_28.insert(cuco::pair{KEY_28[ITEM], ITEM*TB + tid});
BUF_28[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_8_join_build_28[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT>
__global__ void main_14(uint64_t* BUF_29, uint64_t* BUF_IDX_29, HASHTABLE_INSERT HT_29, int64_t* cycles_per_warp_main_14_join_build_29, int64_t* cycles_per_warp_main_14_selection_13, int64_t* cycles_per_warp_main_14_selection_15, int64_t* cycles_per_warp_main_14_selection_16, int64_t* cycles_per_warp_main_14_selection_17, int64_t* cycles_per_warp_main_14_selection_18, DBStringType* part__p_category, DBI32Type* part__p_partkey, size_t part_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_part__p_category[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_category[ITEM] = part__p_category[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_part__p_category[ITEM], "MFGR#14", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_14_selection_13[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_14_selection_15[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_14_selection_16[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_14_selection_17[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_14_selection_18[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_29[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_29[ITEM] = 0;
KEY_29[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_29.insert(cuco::pair{KEY_29[ITEM], ITEM*TB + tid});
BUF_29[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_14_join_build_29[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_30, uint64_t* BUF_IDX_30, HASHTABLE_INSERT HT_30, int64_t* cycles_per_warp_main_1_join_build_30, int64_t* cycles_per_warp_main_1_selection_0, int64_t* cycles_per_warp_main_1_selection_2, int64_t* cycles_per_warp_main_1_selection_3, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_year[ITEM] = date__d_year[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_date__d_year[ITEM], 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year[ITEM], 1998, Predicate::eq));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_3[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_30[ITEMS_PER_THREAD];
DBI32Type reg_date__d_datekey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_datekey[ITEM] = date__d_datekey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_30[ITEM] = 0;
KEY_30[ITEM] |= reg_date__d_datekey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_30.insert(cuco::pair{KEY_30[ITEM], ITEM*TB + tid});
BUF_30[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_build_30[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_31, uint64_t* BUF_IDX_31, HASHTABLE_INSERT HT_31, DBI32Type* customer__c_custkey, size_t customer_size, int64_t* cycles_per_warp_main_5_join_build_31, int64_t* cycles_per_warp_main_5_selection_4, int64_t* cycles_per_warp_main_5_selection_6) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_selection_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_selection_6[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_31[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_31[ITEM] = 0;
KEY_31[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_31.insert(cuco::pair{KEY_31[ITEM], ITEM*TB + tid});
BUF_31[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_join_build_31[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_20(uint64_t* BUF_28, uint64_t* BUF_29, uint64_t* BUF_30, uint64_t* BUF_31, HASHTABLE_PROBE HT_28, HASHTABLE_PROBE HT_29, HASHTABLE_PROBE HT_30, HASHTABLE_PROBE HT_31, HASHTABLE_FIND HT_33, DBI32Type* KEY_33date__d_year, DBI16Type* KEY_33part__p_brand1_encoded, DBI16Type* KEY_33supplier__s_city_encoded, int* SLOT_COUNT_33, DBDecimalType* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_20_aggregation_33, int64_t* cycles_per_warp_main_20_join_probe_28, int64_t* cycles_per_warp_main_20_join_probe_29, int64_t* cycles_per_warp_main_20_join_probe_30, int64_t* cycles_per_warp_main_20_join_probe_31, int64_t* cycles_per_warp_main_20_map_32, int64_t* cycles_per_warp_main_20_selection_19, int64_t* cycles_per_warp_main_20_selection_21, int64_t* cycles_per_warp_main_20_selection_22, int64_t* cycles_per_warp_main_20_selection_23, int64_t* cycles_per_warp_main_20_selection_24, int64_t* cycles_per_warp_main_20_selection_25, int64_t* cycles_per_warp_main_20_selection_26, int64_t* cycles_per_warp_main_20_selection_27, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, DBDecimalType* lineorder__lo_supplycost, size_t lineorder_size, DBI16Type* part__p_brand1_encoded, DBI16Type* supplier__s_city_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_20_selection_19[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_20_selection_21[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_20_selection_22[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_20_selection_23[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_20_selection_24[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_20_selection_25[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_20_selection_26[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_20_selection_27[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_28[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_suppkey[ITEM] = lineorder__lo_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_28[ITEM] = 0;
KEY_28[ITEM] |= reg_lineorder__lo_suppkey[ITEM];
}
//Probe Hash table
int64_t slot_second28[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_28 = HT_28.find(KEY_28[ITEM]);
if (SLOT_28 == HT_28.end()) {selection_flags[ITEM] = 0; continue;}
slot_second28[ITEM] = SLOT_28->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_20_join_probe_28[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_29[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_partkey[ITEM] = lineorder__lo_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_29[ITEM] = 0;
KEY_29[ITEM] |= reg_lineorder__lo_partkey[ITEM];
}
//Probe Hash table
int64_t slot_second29[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_29 = HT_29.find(KEY_29[ITEM]);
if (SLOT_29 == HT_29.end()) {selection_flags[ITEM] = 0; continue;}
slot_second29[ITEM] = SLOT_29->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_20_join_probe_29[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_30[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_orderdate[ITEM] = lineorder__lo_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_30[ITEM] = 0;
KEY_30[ITEM] |= reg_lineorder__lo_orderdate[ITEM];
}
//Probe Hash table
int64_t slot_second30[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_30 = HT_30.find(KEY_30[ITEM]);
if (SLOT_30 == HT_30.end()) {selection_flags[ITEM] = 0; continue;}
slot_second30[ITEM] = SLOT_30->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_20_join_probe_30[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_31[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_custkey[ITEM] = lineorder__lo_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_31[ITEM] = 0;
KEY_31[ITEM] |= reg_lineorder__lo_custkey[ITEM];
}
//Probe Hash table
int64_t slot_second31[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_31 = HT_31.find(KEY_31[ITEM]);
if (SLOT_31 == HT_31.end()) {selection_flags[ITEM] = 0; continue;}
slot_second31[ITEM] = SLOT_31->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_20_join_probe_31[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_20_map_32[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_33[ITEMS_PER_THREAD];
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_date__d_year[ITEM] = date__d_year[BUF_30[slot_second30[ITEM] * 1 + 0]];
}
DBI16Type reg_supplier__s_city_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_supplier__s_city_encoded[ITEM] = supplier__s_city_encoded[BUF_28[slot_second28[ITEM] * 1 + 0]];
}
DBI16Type reg_part__p_brand1_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_brand1_encoded[ITEM] = part__p_brand1_encoded[BUF_29[slot_second29[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_33[ITEM] = 0;
KEY_33[ITEM] |= reg_date__d_year[ITEM];
KEY_33[ITEM] <<= 16;
KEY_33[ITEM] |= reg_supplier__s_city_encoded[ITEM];
KEY_33[ITEM] <<= 16;
KEY_33[ITEM] |= reg_part__p_brand1_encoded[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineorder__lo_supplycost[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_supplycost[ITEM] = lineorder__lo_supplycost[ITEM*TB + tid];
}
DBDecimalType reg_lineorder__lo_revenue[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_revenue[ITEM] = lineorder__lo_revenue[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineorder__lo_revenue[ITEM]) - (reg_lineorder__lo_supplycost[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_33 = get_aggregation_slot(KEY_33[ITEM], HT_33, SLOT_COUNT_33);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_33], reg_map0__tmp_attr1[ITEM]);
KEY_33date__d_year[buf_idx_33] = reg_date__d_year[ITEM];
KEY_33supplier__s_city_encoded[buf_idx_33] = reg_supplier__s_city_encoded[ITEM];
KEY_33part__p_brand1_encoded[buf_idx_33] = reg_part__p_brand1_encoded[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_20_aggregation_33[blockIdx.x] = cycles_per_warp;}
}
__global__ void main_35(size_t COUNT33, DBDecimalType* MAT34aggr0__tmp_attr0, DBI32Type* MAT34date__d_year, DBI16Type* MAT34part__p_brand1_encoded, DBI16Type* MAT34supplier__s_city_encoded, uint64_t* MAT_IDX34, DBDecimalType* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_35_materialize_34, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded, DBI16Type* supplier__s_city_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT33); ++ITEM) {
reg_date__d_year[ITEM] = date__d_year[ITEM*TB + tid];
}
DBI16Type reg_supplier__s_city_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT33); ++ITEM) {
reg_supplier__s_city_encoded[ITEM] = supplier__s_city_encoded[ITEM*TB + tid];
}
DBI16Type reg_part__p_brand1_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT33); ++ITEM) {
reg_part__p_brand1_encoded[ITEM] = part__p_brand1_encoded[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT33); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT33); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx34 = atomicAdd((int*)MAT_IDX34, 1);
MAT34date__d_year[mat_idx34] = reg_date__d_year[ITEM];
MAT34supplier__s_city_encoded[mat_idx34] = reg_supplier__s_city_encoded[ITEM];
MAT34part__p_brand1_encoded[mat_idx34] = reg_part__p_brand1_encoded[ITEM];
MAT34aggr0__tmp_attr0[mat_idx34] = reg_aggr0__tmp_attr0[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_35_materialize_34[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_8_selection_7;
auto main_8_selection_7_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_8_selection_7, sizeof(int64_t) * main_8_selection_7_cpw_size);
hipMemset(d_cycles_per_warp_main_8_selection_7, -1, sizeof(int64_t) * main_8_selection_7_cpw_size);
int64_t* d_cycles_per_warp_main_8_selection_9;
auto main_8_selection_9_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_8_selection_9, sizeof(int64_t) * main_8_selection_9_cpw_size);
hipMemset(d_cycles_per_warp_main_8_selection_9, -1, sizeof(int64_t) * main_8_selection_9_cpw_size);
int64_t* d_cycles_per_warp_main_8_selection_10;
auto main_8_selection_10_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_8_selection_10, sizeof(int64_t) * main_8_selection_10_cpw_size);
hipMemset(d_cycles_per_warp_main_8_selection_10, -1, sizeof(int64_t) * main_8_selection_10_cpw_size);
int64_t* d_cycles_per_warp_main_8_selection_11;
auto main_8_selection_11_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_8_selection_11, sizeof(int64_t) * main_8_selection_11_cpw_size);
hipMemset(d_cycles_per_warp_main_8_selection_11, -1, sizeof(int64_t) * main_8_selection_11_cpw_size);
int64_t* d_cycles_per_warp_main_8_selection_12;
auto main_8_selection_12_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_8_selection_12, sizeof(int64_t) * main_8_selection_12_cpw_size);
hipMemset(d_cycles_per_warp_main_8_selection_12, -1, sizeof(int64_t) * main_8_selection_12_cpw_size);
int64_t* d_cycles_per_warp_main_8_join_build_28;
auto main_8_join_build_28_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_8_join_build_28, sizeof(int64_t) * main_8_join_build_28_cpw_size);
hipMemset(d_cycles_per_warp_main_8_join_build_28, -1, sizeof(int64_t) * main_8_join_build_28_cpw_size);
size_t COUNT28 = supplier_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_28;
hipMalloc(&d_BUF_IDX_28, sizeof(uint64_t));
hipMemset(d_BUF_IDX_28, 0, sizeof(uint64_t));
uint64_t* d_BUF_28;
hipMalloc(&d_BUF_28, sizeof(uint64_t) * COUNT28 * 1);
auto d_HT_28 = cuco::static_map{ (int)COUNT28*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_8<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TB>>>(d_BUF_28, d_BUF_IDX_28, d_HT_28.ref(cuco::insert), d_cycles_per_warp_main_8_join_build_28, d_cycles_per_warp_main_8_selection_10, d_cycles_per_warp_main_8_selection_11, d_cycles_per_warp_main_8_selection_12, d_cycles_per_warp_main_8_selection_7, d_cycles_per_warp_main_8_selection_9, d_supplier__s_nation, d_supplier__s_suppkey, supplier_size);
int64_t* cycles_per_warp_main_8_selection_7 = (int64_t*)malloc(sizeof(int64_t) * main_8_selection_7_cpw_size);
hipMemcpy(cycles_per_warp_main_8_selection_7, d_cycles_per_warp_main_8_selection_7, sizeof(int64_t) * main_8_selection_7_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_8_selection_7 ";
for (auto i=0ull; i < main_8_selection_7_cpw_size; i++) std::cout << cycles_per_warp_main_8_selection_7[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_8_selection_9 = (int64_t*)malloc(sizeof(int64_t) * main_8_selection_9_cpw_size);
hipMemcpy(cycles_per_warp_main_8_selection_9, d_cycles_per_warp_main_8_selection_9, sizeof(int64_t) * main_8_selection_9_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_8_selection_9 ";
for (auto i=0ull; i < main_8_selection_9_cpw_size; i++) std::cout << cycles_per_warp_main_8_selection_9[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_8_selection_10 = (int64_t*)malloc(sizeof(int64_t) * main_8_selection_10_cpw_size);
hipMemcpy(cycles_per_warp_main_8_selection_10, d_cycles_per_warp_main_8_selection_10, sizeof(int64_t) * main_8_selection_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_8_selection_10 ";
for (auto i=0ull; i < main_8_selection_10_cpw_size; i++) std::cout << cycles_per_warp_main_8_selection_10[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_8_selection_11 = (int64_t*)malloc(sizeof(int64_t) * main_8_selection_11_cpw_size);
hipMemcpy(cycles_per_warp_main_8_selection_11, d_cycles_per_warp_main_8_selection_11, sizeof(int64_t) * main_8_selection_11_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_8_selection_11 ";
for (auto i=0ull; i < main_8_selection_11_cpw_size; i++) std::cout << cycles_per_warp_main_8_selection_11[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_8_selection_12 = (int64_t*)malloc(sizeof(int64_t) * main_8_selection_12_cpw_size);
hipMemcpy(cycles_per_warp_main_8_selection_12, d_cycles_per_warp_main_8_selection_12, sizeof(int64_t) * main_8_selection_12_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_8_selection_12 ";
for (auto i=0ull; i < main_8_selection_12_cpw_size; i++) std::cout << cycles_per_warp_main_8_selection_12[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_8_join_build_28 = (int64_t*)malloc(sizeof(int64_t) * main_8_join_build_28_cpw_size);
hipMemcpy(cycles_per_warp_main_8_join_build_28, d_cycles_per_warp_main_8_join_build_28, sizeof(int64_t) * main_8_join_build_28_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_8_join_build_28 ";
for (auto i=0ull; i < main_8_join_build_28_cpw_size; i++) std::cout << cycles_per_warp_main_8_join_build_28[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_14_selection_13;
auto main_14_selection_13_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_14_selection_13, sizeof(int64_t) * main_14_selection_13_cpw_size);
hipMemset(d_cycles_per_warp_main_14_selection_13, -1, sizeof(int64_t) * main_14_selection_13_cpw_size);
int64_t* d_cycles_per_warp_main_14_selection_15;
auto main_14_selection_15_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_14_selection_15, sizeof(int64_t) * main_14_selection_15_cpw_size);
hipMemset(d_cycles_per_warp_main_14_selection_15, -1, sizeof(int64_t) * main_14_selection_15_cpw_size);
int64_t* d_cycles_per_warp_main_14_selection_16;
auto main_14_selection_16_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_14_selection_16, sizeof(int64_t) * main_14_selection_16_cpw_size);
hipMemset(d_cycles_per_warp_main_14_selection_16, -1, sizeof(int64_t) * main_14_selection_16_cpw_size);
int64_t* d_cycles_per_warp_main_14_selection_17;
auto main_14_selection_17_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_14_selection_17, sizeof(int64_t) * main_14_selection_17_cpw_size);
hipMemset(d_cycles_per_warp_main_14_selection_17, -1, sizeof(int64_t) * main_14_selection_17_cpw_size);
int64_t* d_cycles_per_warp_main_14_selection_18;
auto main_14_selection_18_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_14_selection_18, sizeof(int64_t) * main_14_selection_18_cpw_size);
hipMemset(d_cycles_per_warp_main_14_selection_18, -1, sizeof(int64_t) * main_14_selection_18_cpw_size);
int64_t* d_cycles_per_warp_main_14_join_build_29;
auto main_14_join_build_29_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_14_join_build_29, sizeof(int64_t) * main_14_join_build_29_cpw_size);
hipMemset(d_cycles_per_warp_main_14_join_build_29, -1, sizeof(int64_t) * main_14_join_build_29_cpw_size);
size_t COUNT29 = part_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_29;
hipMalloc(&d_BUF_IDX_29, sizeof(uint64_t));
hipMemset(d_BUF_IDX_29, 0, sizeof(uint64_t));
uint64_t* d_BUF_29;
hipMalloc(&d_BUF_29, sizeof(uint64_t) * COUNT29 * 1);
auto d_HT_29 = cuco::static_map{ (int)COUNT29*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_14<<<std::ceil((float)part_size/(float)TILE_SIZE), TB>>>(d_BUF_29, d_BUF_IDX_29, d_HT_29.ref(cuco::insert), d_cycles_per_warp_main_14_join_build_29, d_cycles_per_warp_main_14_selection_13, d_cycles_per_warp_main_14_selection_15, d_cycles_per_warp_main_14_selection_16, d_cycles_per_warp_main_14_selection_17, d_cycles_per_warp_main_14_selection_18, d_part__p_category, d_part__p_partkey, part_size);
int64_t* cycles_per_warp_main_14_selection_13 = (int64_t*)malloc(sizeof(int64_t) * main_14_selection_13_cpw_size);
hipMemcpy(cycles_per_warp_main_14_selection_13, d_cycles_per_warp_main_14_selection_13, sizeof(int64_t) * main_14_selection_13_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_14_selection_13 ";
for (auto i=0ull; i < main_14_selection_13_cpw_size; i++) std::cout << cycles_per_warp_main_14_selection_13[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_14_selection_15 = (int64_t*)malloc(sizeof(int64_t) * main_14_selection_15_cpw_size);
hipMemcpy(cycles_per_warp_main_14_selection_15, d_cycles_per_warp_main_14_selection_15, sizeof(int64_t) * main_14_selection_15_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_14_selection_15 ";
for (auto i=0ull; i < main_14_selection_15_cpw_size; i++) std::cout << cycles_per_warp_main_14_selection_15[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_14_selection_16 = (int64_t*)malloc(sizeof(int64_t) * main_14_selection_16_cpw_size);
hipMemcpy(cycles_per_warp_main_14_selection_16, d_cycles_per_warp_main_14_selection_16, sizeof(int64_t) * main_14_selection_16_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_14_selection_16 ";
for (auto i=0ull; i < main_14_selection_16_cpw_size; i++) std::cout << cycles_per_warp_main_14_selection_16[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_14_selection_17 = (int64_t*)malloc(sizeof(int64_t) * main_14_selection_17_cpw_size);
hipMemcpy(cycles_per_warp_main_14_selection_17, d_cycles_per_warp_main_14_selection_17, sizeof(int64_t) * main_14_selection_17_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_14_selection_17 ";
for (auto i=0ull; i < main_14_selection_17_cpw_size; i++) std::cout << cycles_per_warp_main_14_selection_17[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_14_selection_18 = (int64_t*)malloc(sizeof(int64_t) * main_14_selection_18_cpw_size);
hipMemcpy(cycles_per_warp_main_14_selection_18, d_cycles_per_warp_main_14_selection_18, sizeof(int64_t) * main_14_selection_18_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_14_selection_18 ";
for (auto i=0ull; i < main_14_selection_18_cpw_size; i++) std::cout << cycles_per_warp_main_14_selection_18[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_14_join_build_29 = (int64_t*)malloc(sizeof(int64_t) * main_14_join_build_29_cpw_size);
hipMemcpy(cycles_per_warp_main_14_join_build_29, d_cycles_per_warp_main_14_join_build_29, sizeof(int64_t) * main_14_join_build_29_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_14_join_build_29 ";
for (auto i=0ull; i < main_14_join_build_29_cpw_size; i++) std::cout << cycles_per_warp_main_14_join_build_29[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_2;
auto main_1_selection_2_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_2, -1, sizeof(int64_t) * main_1_selection_2_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_3;
auto main_1_selection_3_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_3, -1, sizeof(int64_t) * main_1_selection_3_cpw_size);
int64_t* d_cycles_per_warp_main_1_join_build_30;
auto main_1_join_build_30_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_build_30, sizeof(int64_t) * main_1_join_build_30_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_build_30, -1, sizeof(int64_t) * main_1_join_build_30_cpw_size);
size_t COUNT30 = date_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_30;
hipMalloc(&d_BUF_IDX_30, sizeof(uint64_t));
hipMemset(d_BUF_IDX_30, 0, sizeof(uint64_t));
uint64_t* d_BUF_30;
hipMalloc(&d_BUF_30, sizeof(uint64_t) * COUNT30 * 1);
auto d_HT_30 = cuco::static_map{ (int)COUNT30*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)date_size/(float)TILE_SIZE), TB>>>(d_BUF_30, d_BUF_IDX_30, d_HT_30.ref(cuco::insert), d_cycles_per_warp_main_1_join_build_30, d_cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_3, d_date__d_datekey, d_date__d_year, date_size);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_2 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_2 ";
for (auto i=0ull; i < main_1_selection_2_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_3 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_3, d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_3 ";
for (auto i=0ull; i < main_1_selection_3_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_3[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_build_30 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_build_30_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_build_30, d_cycles_per_warp_main_1_join_build_30, sizeof(int64_t) * main_1_join_build_30_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_build_30 ";
for (auto i=0ull; i < main_1_join_build_30_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_build_30[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_5_selection_4;
auto main_5_selection_4_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_selection_4, sizeof(int64_t) * main_5_selection_4_cpw_size);
hipMemset(d_cycles_per_warp_main_5_selection_4, -1, sizeof(int64_t) * main_5_selection_4_cpw_size);
int64_t* d_cycles_per_warp_main_5_selection_6;
auto main_5_selection_6_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_selection_6, sizeof(int64_t) * main_5_selection_6_cpw_size);
hipMemset(d_cycles_per_warp_main_5_selection_6, -1, sizeof(int64_t) * main_5_selection_6_cpw_size);
int64_t* d_cycles_per_warp_main_5_join_build_31;
auto main_5_join_build_31_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_join_build_31, sizeof(int64_t) * main_5_join_build_31_cpw_size);
hipMemset(d_cycles_per_warp_main_5_join_build_31, -1, sizeof(int64_t) * main_5_join_build_31_cpw_size);
size_t COUNT31 = customer_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_31;
hipMalloc(&d_BUF_IDX_31, sizeof(uint64_t));
hipMemset(d_BUF_IDX_31, 0, sizeof(uint64_t));
uint64_t* d_BUF_31;
hipMalloc(&d_BUF_31, sizeof(uint64_t) * COUNT31 * 1);
auto d_HT_31 = cuco::static_map{ (int)COUNT31*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)customer_size/(float)TILE_SIZE), TB>>>(d_BUF_31, d_BUF_IDX_31, d_HT_31.ref(cuco::insert), d_customer__c_custkey, customer_size, d_cycles_per_warp_main_5_join_build_31, d_cycles_per_warp_main_5_selection_4, d_cycles_per_warp_main_5_selection_6);
int64_t* cycles_per_warp_main_5_selection_4 = (int64_t*)malloc(sizeof(int64_t) * main_5_selection_4_cpw_size);
hipMemcpy(cycles_per_warp_main_5_selection_4, d_cycles_per_warp_main_5_selection_4, sizeof(int64_t) * main_5_selection_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_selection_4 ";
for (auto i=0ull; i < main_5_selection_4_cpw_size; i++) std::cout << cycles_per_warp_main_5_selection_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_selection_6 = (int64_t*)malloc(sizeof(int64_t) * main_5_selection_6_cpw_size);
hipMemcpy(cycles_per_warp_main_5_selection_6, d_cycles_per_warp_main_5_selection_6, sizeof(int64_t) * main_5_selection_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_selection_6 ";
for (auto i=0ull; i < main_5_selection_6_cpw_size; i++) std::cout << cycles_per_warp_main_5_selection_6[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_join_build_31 = (int64_t*)malloc(sizeof(int64_t) * main_5_join_build_31_cpw_size);
hipMemcpy(cycles_per_warp_main_5_join_build_31, d_cycles_per_warp_main_5_join_build_31, sizeof(int64_t) * main_5_join_build_31_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_join_build_31 ";
for (auto i=0ull; i < main_5_join_build_31_cpw_size; i++) std::cout << cycles_per_warp_main_5_join_build_31[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_20_selection_19;
auto main_20_selection_19_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_20_selection_19, sizeof(int64_t) * main_20_selection_19_cpw_size);
hipMemset(d_cycles_per_warp_main_20_selection_19, -1, sizeof(int64_t) * main_20_selection_19_cpw_size);
int64_t* d_cycles_per_warp_main_20_selection_21;
auto main_20_selection_21_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_20_selection_21, sizeof(int64_t) * main_20_selection_21_cpw_size);
hipMemset(d_cycles_per_warp_main_20_selection_21, -1, sizeof(int64_t) * main_20_selection_21_cpw_size);
int64_t* d_cycles_per_warp_main_20_selection_22;
auto main_20_selection_22_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_20_selection_22, sizeof(int64_t) * main_20_selection_22_cpw_size);
hipMemset(d_cycles_per_warp_main_20_selection_22, -1, sizeof(int64_t) * main_20_selection_22_cpw_size);
int64_t* d_cycles_per_warp_main_20_selection_23;
auto main_20_selection_23_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_20_selection_23, sizeof(int64_t) * main_20_selection_23_cpw_size);
hipMemset(d_cycles_per_warp_main_20_selection_23, -1, sizeof(int64_t) * main_20_selection_23_cpw_size);
int64_t* d_cycles_per_warp_main_20_selection_24;
auto main_20_selection_24_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_20_selection_24, sizeof(int64_t) * main_20_selection_24_cpw_size);
hipMemset(d_cycles_per_warp_main_20_selection_24, -1, sizeof(int64_t) * main_20_selection_24_cpw_size);
int64_t* d_cycles_per_warp_main_20_selection_25;
auto main_20_selection_25_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_20_selection_25, sizeof(int64_t) * main_20_selection_25_cpw_size);
hipMemset(d_cycles_per_warp_main_20_selection_25, -1, sizeof(int64_t) * main_20_selection_25_cpw_size);
int64_t* d_cycles_per_warp_main_20_selection_26;
auto main_20_selection_26_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_20_selection_26, sizeof(int64_t) * main_20_selection_26_cpw_size);
hipMemset(d_cycles_per_warp_main_20_selection_26, -1, sizeof(int64_t) * main_20_selection_26_cpw_size);
int64_t* d_cycles_per_warp_main_20_selection_27;
auto main_20_selection_27_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_20_selection_27, sizeof(int64_t) * main_20_selection_27_cpw_size);
hipMemset(d_cycles_per_warp_main_20_selection_27, -1, sizeof(int64_t) * main_20_selection_27_cpw_size);
int64_t* d_cycles_per_warp_main_20_join_probe_28;
auto main_20_join_probe_28_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_20_join_probe_28, sizeof(int64_t) * main_20_join_probe_28_cpw_size);
hipMemset(d_cycles_per_warp_main_20_join_probe_28, -1, sizeof(int64_t) * main_20_join_probe_28_cpw_size);
int64_t* d_cycles_per_warp_main_20_join_probe_29;
auto main_20_join_probe_29_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_20_join_probe_29, sizeof(int64_t) * main_20_join_probe_29_cpw_size);
hipMemset(d_cycles_per_warp_main_20_join_probe_29, -1, sizeof(int64_t) * main_20_join_probe_29_cpw_size);
int64_t* d_cycles_per_warp_main_20_join_probe_30;
auto main_20_join_probe_30_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_20_join_probe_30, sizeof(int64_t) * main_20_join_probe_30_cpw_size);
hipMemset(d_cycles_per_warp_main_20_join_probe_30, -1, sizeof(int64_t) * main_20_join_probe_30_cpw_size);
int64_t* d_cycles_per_warp_main_20_join_probe_31;
auto main_20_join_probe_31_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_20_join_probe_31, sizeof(int64_t) * main_20_join_probe_31_cpw_size);
hipMemset(d_cycles_per_warp_main_20_join_probe_31, -1, sizeof(int64_t) * main_20_join_probe_31_cpw_size);
int64_t* d_cycles_per_warp_main_20_map_32;
auto main_20_map_32_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_20_map_32, sizeof(int64_t) * main_20_map_32_cpw_size);
hipMemset(d_cycles_per_warp_main_20_map_32, -1, sizeof(int64_t) * main_20_map_32_cpw_size);
int64_t* d_cycles_per_warp_main_20_aggregation_33;
auto main_20_aggregation_33_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_20_aggregation_33, sizeof(int64_t) * main_20_aggregation_33_cpw_size);
hipMemset(d_cycles_per_warp_main_20_aggregation_33, -1, sizeof(int64_t) * main_20_aggregation_33_cpw_size);
size_t COUNT33 = 2259;
auto d_HT_33 = cuco::static_map{ (int)2259*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_33;
hipMalloc(&d_SLOT_COUNT_33, sizeof(int));
hipMemset(d_SLOT_COUNT_33, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT33);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT33);
DBI32Type* d_KEY_33date__d_year;
hipMalloc(&d_KEY_33date__d_year, sizeof(DBI32Type) * COUNT33);
hipMemset(d_KEY_33date__d_year, 0, sizeof(DBI32Type) * COUNT33);
DBI16Type* d_KEY_33supplier__s_city_encoded;
hipMalloc(&d_KEY_33supplier__s_city_encoded, sizeof(DBI16Type) * COUNT33);
hipMemset(d_KEY_33supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT33);
DBI16Type* d_KEY_33part__p_brand1_encoded;
hipMalloc(&d_KEY_33part__p_brand1_encoded, sizeof(DBI16Type) * COUNT33);
hipMemset(d_KEY_33part__p_brand1_encoded, 0, sizeof(DBI16Type) * COUNT33);
main_20<<<std::ceil((float)lineorder_size/(float)TILE_SIZE), TB>>>(d_BUF_28, d_BUF_29, d_BUF_30, d_BUF_31, d_HT_28.ref(cuco::find), d_HT_29.ref(cuco::find), d_HT_30.ref(cuco::find), d_HT_31.ref(cuco::find), d_HT_33.ref(cuco::insert_and_find), d_KEY_33date__d_year, d_KEY_33part__p_brand1_encoded, d_KEY_33supplier__s_city_encoded, d_SLOT_COUNT_33, d_aggr0__tmp_attr0, d_cycles_per_warp_main_20_aggregation_33, d_cycles_per_warp_main_20_join_probe_28, d_cycles_per_warp_main_20_join_probe_29, d_cycles_per_warp_main_20_join_probe_30, d_cycles_per_warp_main_20_join_probe_31, d_cycles_per_warp_main_20_map_32, d_cycles_per_warp_main_20_selection_19, d_cycles_per_warp_main_20_selection_21, d_cycles_per_warp_main_20_selection_22, d_cycles_per_warp_main_20_selection_23, d_cycles_per_warp_main_20_selection_24, d_cycles_per_warp_main_20_selection_25, d_cycles_per_warp_main_20_selection_26, d_cycles_per_warp_main_20_selection_27, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, d_lineorder__lo_supplycost, lineorder_size, d_part__p_brand1_encoded, d_supplier__s_city_encoded);
int64_t* cycles_per_warp_main_20_selection_19 = (int64_t*)malloc(sizeof(int64_t) * main_20_selection_19_cpw_size);
hipMemcpy(cycles_per_warp_main_20_selection_19, d_cycles_per_warp_main_20_selection_19, sizeof(int64_t) * main_20_selection_19_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_20_selection_19 ";
for (auto i=0ull; i < main_20_selection_19_cpw_size; i++) std::cout << cycles_per_warp_main_20_selection_19[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_20_selection_21 = (int64_t*)malloc(sizeof(int64_t) * main_20_selection_21_cpw_size);
hipMemcpy(cycles_per_warp_main_20_selection_21, d_cycles_per_warp_main_20_selection_21, sizeof(int64_t) * main_20_selection_21_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_20_selection_21 ";
for (auto i=0ull; i < main_20_selection_21_cpw_size; i++) std::cout << cycles_per_warp_main_20_selection_21[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_20_selection_22 = (int64_t*)malloc(sizeof(int64_t) * main_20_selection_22_cpw_size);
hipMemcpy(cycles_per_warp_main_20_selection_22, d_cycles_per_warp_main_20_selection_22, sizeof(int64_t) * main_20_selection_22_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_20_selection_22 ";
for (auto i=0ull; i < main_20_selection_22_cpw_size; i++) std::cout << cycles_per_warp_main_20_selection_22[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_20_selection_23 = (int64_t*)malloc(sizeof(int64_t) * main_20_selection_23_cpw_size);
hipMemcpy(cycles_per_warp_main_20_selection_23, d_cycles_per_warp_main_20_selection_23, sizeof(int64_t) * main_20_selection_23_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_20_selection_23 ";
for (auto i=0ull; i < main_20_selection_23_cpw_size; i++) std::cout << cycles_per_warp_main_20_selection_23[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_20_selection_24 = (int64_t*)malloc(sizeof(int64_t) * main_20_selection_24_cpw_size);
hipMemcpy(cycles_per_warp_main_20_selection_24, d_cycles_per_warp_main_20_selection_24, sizeof(int64_t) * main_20_selection_24_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_20_selection_24 ";
for (auto i=0ull; i < main_20_selection_24_cpw_size; i++) std::cout << cycles_per_warp_main_20_selection_24[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_20_selection_25 = (int64_t*)malloc(sizeof(int64_t) * main_20_selection_25_cpw_size);
hipMemcpy(cycles_per_warp_main_20_selection_25, d_cycles_per_warp_main_20_selection_25, sizeof(int64_t) * main_20_selection_25_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_20_selection_25 ";
for (auto i=0ull; i < main_20_selection_25_cpw_size; i++) std::cout << cycles_per_warp_main_20_selection_25[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_20_selection_26 = (int64_t*)malloc(sizeof(int64_t) * main_20_selection_26_cpw_size);
hipMemcpy(cycles_per_warp_main_20_selection_26, d_cycles_per_warp_main_20_selection_26, sizeof(int64_t) * main_20_selection_26_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_20_selection_26 ";
for (auto i=0ull; i < main_20_selection_26_cpw_size; i++) std::cout << cycles_per_warp_main_20_selection_26[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_20_selection_27 = (int64_t*)malloc(sizeof(int64_t) * main_20_selection_27_cpw_size);
hipMemcpy(cycles_per_warp_main_20_selection_27, d_cycles_per_warp_main_20_selection_27, sizeof(int64_t) * main_20_selection_27_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_20_selection_27 ";
for (auto i=0ull; i < main_20_selection_27_cpw_size; i++) std::cout << cycles_per_warp_main_20_selection_27[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_20_join_probe_28 = (int64_t*)malloc(sizeof(int64_t) * main_20_join_probe_28_cpw_size);
hipMemcpy(cycles_per_warp_main_20_join_probe_28, d_cycles_per_warp_main_20_join_probe_28, sizeof(int64_t) * main_20_join_probe_28_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_20_join_probe_28 ";
for (auto i=0ull; i < main_20_join_probe_28_cpw_size; i++) std::cout << cycles_per_warp_main_20_join_probe_28[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_20_join_probe_29 = (int64_t*)malloc(sizeof(int64_t) * main_20_join_probe_29_cpw_size);
hipMemcpy(cycles_per_warp_main_20_join_probe_29, d_cycles_per_warp_main_20_join_probe_29, sizeof(int64_t) * main_20_join_probe_29_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_20_join_probe_29 ";
for (auto i=0ull; i < main_20_join_probe_29_cpw_size; i++) std::cout << cycles_per_warp_main_20_join_probe_29[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_20_join_probe_30 = (int64_t*)malloc(sizeof(int64_t) * main_20_join_probe_30_cpw_size);
hipMemcpy(cycles_per_warp_main_20_join_probe_30, d_cycles_per_warp_main_20_join_probe_30, sizeof(int64_t) * main_20_join_probe_30_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_20_join_probe_30 ";
for (auto i=0ull; i < main_20_join_probe_30_cpw_size; i++) std::cout << cycles_per_warp_main_20_join_probe_30[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_20_join_probe_31 = (int64_t*)malloc(sizeof(int64_t) * main_20_join_probe_31_cpw_size);
hipMemcpy(cycles_per_warp_main_20_join_probe_31, d_cycles_per_warp_main_20_join_probe_31, sizeof(int64_t) * main_20_join_probe_31_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_20_join_probe_31 ";
for (auto i=0ull; i < main_20_join_probe_31_cpw_size; i++) std::cout << cycles_per_warp_main_20_join_probe_31[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_20_map_32 = (int64_t*)malloc(sizeof(int64_t) * main_20_map_32_cpw_size);
hipMemcpy(cycles_per_warp_main_20_map_32, d_cycles_per_warp_main_20_map_32, sizeof(int64_t) * main_20_map_32_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_20_map_32 ";
for (auto i=0ull; i < main_20_map_32_cpw_size; i++) std::cout << cycles_per_warp_main_20_map_32[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_20_aggregation_33 = (int64_t*)malloc(sizeof(int64_t) * main_20_aggregation_33_cpw_size);
hipMemcpy(cycles_per_warp_main_20_aggregation_33, d_cycles_per_warp_main_20_aggregation_33, sizeof(int64_t) * main_20_aggregation_33_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_20_aggregation_33 ";
for (auto i=0ull; i < main_20_aggregation_33_cpw_size; i++) std::cout << cycles_per_warp_main_20_aggregation_33[i] << " ";
std::cout << std::endl;
COUNT33 = d_HT_33.size();
int64_t* d_cycles_per_warp_main_35_materialize_34;
auto main_35_materialize_34_cpw_size = std::ceil((float)COUNT33/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_35_materialize_34, sizeof(int64_t) * main_35_materialize_34_cpw_size);
hipMemset(d_cycles_per_warp_main_35_materialize_34, -1, sizeof(int64_t) * main_35_materialize_34_cpw_size);
size_t COUNT34 = COUNT33;
//Materialize buffers
uint64_t* d_MAT_IDX34;
hipMalloc(&d_MAT_IDX34, sizeof(uint64_t));
hipMemset(d_MAT_IDX34, 0, sizeof(uint64_t));
auto MAT34date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT34);
DBI32Type* d_MAT34date__d_year;
hipMalloc(&d_MAT34date__d_year, sizeof(DBI32Type) * COUNT34);
auto MAT34supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT34);
DBI16Type* d_MAT34supplier__s_city_encoded;
hipMalloc(&d_MAT34supplier__s_city_encoded, sizeof(DBI16Type) * COUNT34);
auto MAT34part__p_brand1_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT34);
DBI16Type* d_MAT34part__p_brand1_encoded;
hipMalloc(&d_MAT34part__p_brand1_encoded, sizeof(DBI16Type) * COUNT34);
auto MAT34aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT34);
DBDecimalType* d_MAT34aggr0__tmp_attr0;
hipMalloc(&d_MAT34aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT34);
main_35<<<std::ceil((float)COUNT33/(float)TILE_SIZE), TB>>>(COUNT33, d_MAT34aggr0__tmp_attr0, d_MAT34date__d_year, d_MAT34part__p_brand1_encoded, d_MAT34supplier__s_city_encoded, d_MAT_IDX34, d_aggr0__tmp_attr0, d_cycles_per_warp_main_35_materialize_34, d_KEY_33date__d_year, d_KEY_33part__p_brand1_encoded, d_KEY_33supplier__s_city_encoded);
uint64_t MATCOUNT_34 = 0;
hipMemcpy(&MATCOUNT_34, d_MAT_IDX34, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT34date__d_year, d_MAT34date__d_year, sizeof(DBI32Type) * COUNT34, hipMemcpyDeviceToHost);
hipMemcpy(MAT34supplier__s_city_encoded, d_MAT34supplier__s_city_encoded, sizeof(DBI16Type) * COUNT34, hipMemcpyDeviceToHost);
hipMemcpy(MAT34part__p_brand1_encoded, d_MAT34part__p_brand1_encoded, sizeof(DBI16Type) * COUNT34, hipMemcpyDeviceToHost);
hipMemcpy(MAT34aggr0__tmp_attr0, d_MAT34aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT34, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_35_materialize_34 = (int64_t*)malloc(sizeof(int64_t) * main_35_materialize_34_cpw_size);
hipMemcpy(cycles_per_warp_main_35_materialize_34, d_cycles_per_warp_main_35_materialize_34, sizeof(int64_t) * main_35_materialize_34_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_35_materialize_34 ";
for (auto i=0ull; i < main_35_materialize_34_cpw_size; i++) std::cout << cycles_per_warp_main_35_materialize_34[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_28);
hipFree(d_BUF_IDX_28);
hipFree(d_BUF_29);
hipFree(d_BUF_IDX_29);
hipFree(d_BUF_30);
hipFree(d_BUF_IDX_30);
hipFree(d_BUF_31);
hipFree(d_BUF_IDX_31);
hipFree(d_KEY_33date__d_year);
hipFree(d_KEY_33part__p_brand1_encoded);
hipFree(d_KEY_33supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_MAT34aggr0__tmp_attr0);
hipFree(d_MAT34date__d_year);
hipFree(d_MAT34part__p_brand1_encoded);
hipFree(d_MAT34supplier__s_city_encoded);
hipFree(d_MAT_IDX34);
free(MAT34aggr0__tmp_attr0);
free(MAT34date__d_year);
free(MAT34part__p_brand1_encoded);
free(MAT34supplier__s_city_encoded);
}