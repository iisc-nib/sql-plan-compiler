#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5d0dcd849740(uint64_t* COUNT5d0dcd85fc60, DBStringType* customer__c_city, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_city = customer__c_city[tid];
if (!((evaluatePredicate(reg_customer__c_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_customer__c_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5d0dcd85fc60, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5d0dcd849740(uint64_t* BUF_5d0dcd85fc60, uint64_t* BUF_IDX_5d0dcd85fc60, HASHTABLE_INSERT HT_5d0dcd85fc60, DBStringType* customer__c_city, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_city = customer__c_city[tid];
if (!((evaluatePredicate(reg_customer__c_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_customer__c_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5d0dcd85fc60 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5d0dcd85fc60 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_5d0dcd85fc60 = atomicAdd((int*)BUF_IDX_5d0dcd85fc60, 1);
HT_5d0dcd85fc60.insert(cuco::pair{KEY_5d0dcd85fc60, buf_idx_5d0dcd85fc60});
BUF_5d0dcd85fc60[buf_idx_5d0dcd85fc60 * 1 + 0] = tid;
}
__global__ void count_5d0dcd849ce0(uint64_t* COUNT5d0dcd8686e0, DBStringType* supplier__s_city, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_city = supplier__s_city[tid];
if (!((evaluatePredicate(reg_supplier__s_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_supplier__s_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5d0dcd8686e0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5d0dcd849ce0(uint64_t* BUF_5d0dcd8686e0, uint64_t* BUF_IDX_5d0dcd8686e0, HASHTABLE_INSERT HT_5d0dcd8686e0, DBStringType* supplier__s_city, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_city = supplier__s_city[tid];
if (!((evaluatePredicate(reg_supplier__s_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_supplier__s_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5d0dcd8686e0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5d0dcd8686e0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5d0dcd8686e0 = atomicAdd((int*)BUF_IDX_5d0dcd8686e0, 1);
HT_5d0dcd8686e0.insert(cuco::pair{KEY_5d0dcd8686e0, buf_idx_5d0dcd8686e0});
BUF_5d0dcd8686e0[buf_idx_5d0dcd8686e0 * 1 + 0] = tid;
}
__global__ void count_5d0dcd875f00(uint64_t* COUNT5d0dcd860aa0, DBStringType* date__d_yearmonth, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_yearmonth = date__d_yearmonth[tid];
if (!(evaluatePredicate(reg_date__d_yearmonth, "Dec1997", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5d0dcd860aa0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5d0dcd875f00(uint64_t* BUF_5d0dcd860aa0, uint64_t* BUF_IDX_5d0dcd860aa0, HASHTABLE_INSERT HT_5d0dcd860aa0, DBI32Type* date__d_datekey, DBStringType* date__d_yearmonth, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_yearmonth = date__d_yearmonth[tid];
if (!(evaluatePredicate(reg_date__d_yearmonth, "Dec1997", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5d0dcd860aa0 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5d0dcd860aa0 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_5d0dcd860aa0 = atomicAdd((int*)BUF_IDX_5d0dcd860aa0, 1);
HT_5d0dcd860aa0.insert(cuco::pair{KEY_5d0dcd860aa0, buf_idx_5d0dcd860aa0});
BUF_5d0dcd860aa0[buf_idx_5d0dcd860aa0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5d0dcd867db0(uint64_t* BUF_5d0dcd85fc60, uint64_t* BUF_5d0dcd860aa0, uint64_t* BUF_5d0dcd8686e0, HASHTABLE_INSERT HT_5d0dcd819160, HASHTABLE_PROBE HT_5d0dcd85fc60, HASHTABLE_PROBE HT_5d0dcd860aa0, HASHTABLE_PROBE HT_5d0dcd8686e0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5d0dcd85fc60 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5d0dcd85fc60 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_5d0dcd85fc60.for_each(KEY_5d0dcd85fc60, [&] __device__ (auto const SLOT_5d0dcd85fc60) {

auto const [slot_first5d0dcd85fc60, slot_second5d0dcd85fc60] = SLOT_5d0dcd85fc60;
if (!(true)) return;
uint64_t KEY_5d0dcd8686e0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5d0dcd8686e0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5d0dcd8686e0.for_each(KEY_5d0dcd8686e0, [&] __device__ (auto const SLOT_5d0dcd8686e0) {

auto const [slot_first5d0dcd8686e0, slot_second5d0dcd8686e0] = SLOT_5d0dcd8686e0;
if (!(true)) return;
uint64_t KEY_5d0dcd860aa0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5d0dcd860aa0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5d0dcd860aa0.for_each(KEY_5d0dcd860aa0, [&] __device__ (auto const SLOT_5d0dcd860aa0) {

auto const [slot_first5d0dcd860aa0, slot_second5d0dcd860aa0] = SLOT_5d0dcd860aa0;
if (!(true)) return;
uint64_t KEY_5d0dcd819160 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_5d0dcd85fc60[slot_second5d0dcd85fc60 * 1 + 0]];

KEY_5d0dcd819160 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_5d0dcd8686e0[slot_second5d0dcd8686e0 * 1 + 0]];
KEY_5d0dcd819160 <<= 16;
KEY_5d0dcd819160 |= reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[BUF_5d0dcd860aa0[slot_second5d0dcd860aa0 * 1 + 0]];
KEY_5d0dcd819160 <<= 32;
KEY_5d0dcd819160 |= reg_date__d_year;
//Create aggregation hash table
HT_5d0dcd819160.insert(cuco::pair{KEY_5d0dcd819160, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5d0dcd867db0(uint64_t* BUF_5d0dcd85fc60, uint64_t* BUF_5d0dcd860aa0, uint64_t* BUF_5d0dcd8686e0, HASHTABLE_FIND HT_5d0dcd819160, HASHTABLE_PROBE HT_5d0dcd85fc60, HASHTABLE_PROBE HT_5d0dcd860aa0, HASHTABLE_PROBE HT_5d0dcd8686e0, DBI16Type* KEY_5d0dcd819160customer__c_city_encoded, DBI32Type* KEY_5d0dcd819160date__d_year, DBI16Type* KEY_5d0dcd819160supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5d0dcd85fc60 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5d0dcd85fc60 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_5d0dcd85fc60.for_each(KEY_5d0dcd85fc60, [&] __device__ (auto const SLOT_5d0dcd85fc60) {
auto const [slot_first5d0dcd85fc60, slot_second5d0dcd85fc60] = SLOT_5d0dcd85fc60;
if (!(true)) return;
uint64_t KEY_5d0dcd8686e0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5d0dcd8686e0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5d0dcd8686e0.for_each(KEY_5d0dcd8686e0, [&] __device__ (auto const SLOT_5d0dcd8686e0) {
auto const [slot_first5d0dcd8686e0, slot_second5d0dcd8686e0] = SLOT_5d0dcd8686e0;
if (!(true)) return;
uint64_t KEY_5d0dcd860aa0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5d0dcd860aa0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5d0dcd860aa0.for_each(KEY_5d0dcd860aa0, [&] __device__ (auto const SLOT_5d0dcd860aa0) {
auto const [slot_first5d0dcd860aa0, slot_second5d0dcd860aa0] = SLOT_5d0dcd860aa0;
if (!(true)) return;
uint64_t KEY_5d0dcd819160 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_5d0dcd85fc60[slot_second5d0dcd85fc60 * 1 + 0]];

KEY_5d0dcd819160 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_5d0dcd8686e0[slot_second5d0dcd8686e0 * 1 + 0]];
KEY_5d0dcd819160 <<= 16;
KEY_5d0dcd819160 |= reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[BUF_5d0dcd860aa0[slot_second5d0dcd860aa0 * 1 + 0]];
KEY_5d0dcd819160 <<= 32;
KEY_5d0dcd819160 |= reg_date__d_year;
//Aggregate in hashtable
auto buf_idx_5d0dcd819160 = HT_5d0dcd819160.find(KEY_5d0dcd819160)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5d0dcd819160], reg_lineorder__lo_revenue);
KEY_5d0dcd819160customer__c_city_encoded[buf_idx_5d0dcd819160] = reg_customer__c_city_encoded;
KEY_5d0dcd819160supplier__s_city_encoded[buf_idx_5d0dcd819160] = reg_supplier__s_city_encoded;
KEY_5d0dcd819160date__d_year[buf_idx_5d0dcd819160] = reg_date__d_year;
});
});
});
}
__global__ void count_5d0dcd883400(size_t COUNT5d0dcd819160, uint64_t* COUNT5d0dcd82c7f0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5d0dcd819160) return;
//Materialize count
atomicAdd((int*)COUNT5d0dcd82c7f0, 1);
}
__global__ void main_5d0dcd883400(size_t COUNT5d0dcd819160, DBDecimalType* MAT5d0dcd82c7f0aggr0__tmp_attr0, DBI16Type* MAT5d0dcd82c7f0customer__c_city_encoded, DBI32Type* MAT5d0dcd82c7f0date__d_year, DBI16Type* MAT5d0dcd82c7f0supplier__s_city_encoded, uint64_t* MAT_IDX5d0dcd82c7f0, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5d0dcd819160) return;
//Materialize buffers
auto mat_idx5d0dcd82c7f0 = atomicAdd((int*)MAT_IDX5d0dcd82c7f0, 1);
auto reg_customer__c_city_encoded = customer__c_city_encoded[tid];
MAT5d0dcd82c7f0customer__c_city_encoded[mat_idx5d0dcd82c7f0] = reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[tid];
MAT5d0dcd82c7f0supplier__s_city_encoded[mat_idx5d0dcd82c7f0] = reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[tid];
MAT5d0dcd82c7f0date__d_year[mat_idx5d0dcd82c7f0] = reg_date__d_year;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5d0dcd82c7f0aggr0__tmp_attr0[mat_idx5d0dcd82c7f0] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT5d0dcd85fc60;
hipMalloc(&d_COUNT5d0dcd85fc60, sizeof(uint64_t));
hipMemset(d_COUNT5d0dcd85fc60, 0, sizeof(uint64_t));
count_5d0dcd849740<<<std::ceil((float)customer_size/128.), 128>>>(d_COUNT5d0dcd85fc60, d_customer__c_city, customer_size);
uint64_t COUNT5d0dcd85fc60;
hipMemcpy(&COUNT5d0dcd85fc60, d_COUNT5d0dcd85fc60, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5d0dcd85fc60;
hipMalloc(&d_BUF_IDX_5d0dcd85fc60, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5d0dcd85fc60, 0, sizeof(uint64_t));
uint64_t* d_BUF_5d0dcd85fc60;
hipMalloc(&d_BUF_5d0dcd85fc60, sizeof(uint64_t) * COUNT5d0dcd85fc60 * 1);
auto d_HT_5d0dcd85fc60 = cuco::experimental::static_multimap{ (int)COUNT5d0dcd85fc60*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5d0dcd849740<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_5d0dcd85fc60, d_BUF_IDX_5d0dcd85fc60, d_HT_5d0dcd85fc60.ref(cuco::insert), d_customer__c_city, d_customer__c_custkey, customer_size);
//Materialize count
uint64_t* d_COUNT5d0dcd8686e0;
hipMalloc(&d_COUNT5d0dcd8686e0, sizeof(uint64_t));
hipMemset(d_COUNT5d0dcd8686e0, 0, sizeof(uint64_t));
count_5d0dcd849ce0<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT5d0dcd8686e0, d_supplier__s_city, supplier_size);
uint64_t COUNT5d0dcd8686e0;
hipMemcpy(&COUNT5d0dcd8686e0, d_COUNT5d0dcd8686e0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5d0dcd8686e0;
hipMalloc(&d_BUF_IDX_5d0dcd8686e0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5d0dcd8686e0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5d0dcd8686e0;
hipMalloc(&d_BUF_5d0dcd8686e0, sizeof(uint64_t) * COUNT5d0dcd8686e0 * 1);
auto d_HT_5d0dcd8686e0 = cuco::experimental::static_multimap{ (int)COUNT5d0dcd8686e0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5d0dcd849ce0<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_5d0dcd8686e0, d_BUF_IDX_5d0dcd8686e0, d_HT_5d0dcd8686e0.ref(cuco::insert), d_supplier__s_city, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5d0dcd860aa0;
hipMalloc(&d_COUNT5d0dcd860aa0, sizeof(uint64_t));
hipMemset(d_COUNT5d0dcd860aa0, 0, sizeof(uint64_t));
count_5d0dcd875f00<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT5d0dcd860aa0, d_date__d_yearmonth, date_size);
uint64_t COUNT5d0dcd860aa0;
hipMemcpy(&COUNT5d0dcd860aa0, d_COUNT5d0dcd860aa0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5d0dcd860aa0;
hipMalloc(&d_BUF_IDX_5d0dcd860aa0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5d0dcd860aa0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5d0dcd860aa0;
hipMalloc(&d_BUF_5d0dcd860aa0, sizeof(uint64_t) * COUNT5d0dcd860aa0 * 1);
auto d_HT_5d0dcd860aa0 = cuco::experimental::static_multimap{ (int)COUNT5d0dcd860aa0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5d0dcd875f00<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_5d0dcd860aa0, d_BUF_IDX_5d0dcd860aa0, d_HT_5d0dcd860aa0.ref(cuco::insert), d_date__d_datekey, d_date__d_yearmonth, date_size);
//Create aggregation hash table
auto d_HT_5d0dcd819160 = cuco::static_map{ (int)3*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5d0dcd867db0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5d0dcd85fc60, d_BUF_5d0dcd860aa0, d_BUF_5d0dcd8686e0, d_HT_5d0dcd819160.ref(cuco::insert), d_HT_5d0dcd85fc60.ref(cuco::for_each), d_HT_5d0dcd860aa0.ref(cuco::for_each), d_HT_5d0dcd8686e0.ref(cuco::for_each), d_customer__c_city_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
size_t COUNT5d0dcd819160 = d_HT_5d0dcd819160.size();
thrust::device_vector<int64_t> keys_5d0dcd819160(COUNT5d0dcd819160), vals_5d0dcd819160(COUNT5d0dcd819160);
d_HT_5d0dcd819160.retrieve_all(keys_5d0dcd819160.begin(), vals_5d0dcd819160.begin());
d_HT_5d0dcd819160.clear();
int64_t* raw_keys5d0dcd819160 = thrust::raw_pointer_cast(keys_5d0dcd819160.data());
insertKeys<<<std::ceil((float)COUNT5d0dcd819160/128.), 128>>>(raw_keys5d0dcd819160, d_HT_5d0dcd819160.ref(cuco::insert), COUNT5d0dcd819160);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5d0dcd819160);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5d0dcd819160);
DBI16Type* d_KEY_5d0dcd819160customer__c_city_encoded;
hipMalloc(&d_KEY_5d0dcd819160customer__c_city_encoded, sizeof(DBI16Type) * COUNT5d0dcd819160);
hipMemset(d_KEY_5d0dcd819160customer__c_city_encoded, 0, sizeof(DBI16Type) * COUNT5d0dcd819160);
DBI16Type* d_KEY_5d0dcd819160supplier__s_city_encoded;
hipMalloc(&d_KEY_5d0dcd819160supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5d0dcd819160);
hipMemset(d_KEY_5d0dcd819160supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT5d0dcd819160);
DBI32Type* d_KEY_5d0dcd819160date__d_year;
hipMalloc(&d_KEY_5d0dcd819160date__d_year, sizeof(DBI32Type) * COUNT5d0dcd819160);
hipMemset(d_KEY_5d0dcd819160date__d_year, 0, sizeof(DBI32Type) * COUNT5d0dcd819160);
main_5d0dcd867db0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5d0dcd85fc60, d_BUF_5d0dcd860aa0, d_BUF_5d0dcd8686e0, d_HT_5d0dcd819160.ref(cuco::find), d_HT_5d0dcd85fc60.ref(cuco::for_each), d_HT_5d0dcd860aa0.ref(cuco::for_each), d_HT_5d0dcd8686e0.ref(cuco::for_each), d_KEY_5d0dcd819160customer__c_city_encoded, d_KEY_5d0dcd819160date__d_year, d_KEY_5d0dcd819160supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_city_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
//Materialize count
uint64_t* d_COUNT5d0dcd82c7f0;
hipMalloc(&d_COUNT5d0dcd82c7f0, sizeof(uint64_t));
hipMemset(d_COUNT5d0dcd82c7f0, 0, sizeof(uint64_t));
count_5d0dcd883400<<<std::ceil((float)COUNT5d0dcd819160/128.), 128>>>(COUNT5d0dcd819160, d_COUNT5d0dcd82c7f0);
uint64_t COUNT5d0dcd82c7f0;
hipMemcpy(&COUNT5d0dcd82c7f0, d_COUNT5d0dcd82c7f0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5d0dcd82c7f0;
hipMalloc(&d_MAT_IDX5d0dcd82c7f0, sizeof(uint64_t));
hipMemset(d_MAT_IDX5d0dcd82c7f0, 0, sizeof(uint64_t));
auto MAT5d0dcd82c7f0customer__c_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5d0dcd82c7f0);
DBI16Type* d_MAT5d0dcd82c7f0customer__c_city_encoded;
hipMalloc(&d_MAT5d0dcd82c7f0customer__c_city_encoded, sizeof(DBI16Type) * COUNT5d0dcd82c7f0);
auto MAT5d0dcd82c7f0supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5d0dcd82c7f0);
DBI16Type* d_MAT5d0dcd82c7f0supplier__s_city_encoded;
hipMalloc(&d_MAT5d0dcd82c7f0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5d0dcd82c7f0);
auto MAT5d0dcd82c7f0date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5d0dcd82c7f0);
DBI32Type* d_MAT5d0dcd82c7f0date__d_year;
hipMalloc(&d_MAT5d0dcd82c7f0date__d_year, sizeof(DBI32Type) * COUNT5d0dcd82c7f0);
auto MAT5d0dcd82c7f0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5d0dcd82c7f0);
DBDecimalType* d_MAT5d0dcd82c7f0aggr0__tmp_attr0;
hipMalloc(&d_MAT5d0dcd82c7f0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5d0dcd82c7f0);
main_5d0dcd883400<<<std::ceil((float)COUNT5d0dcd819160/128.), 128>>>(COUNT5d0dcd819160, d_MAT5d0dcd82c7f0aggr0__tmp_attr0, d_MAT5d0dcd82c7f0customer__c_city_encoded, d_MAT5d0dcd82c7f0date__d_year, d_MAT5d0dcd82c7f0supplier__s_city_encoded, d_MAT_IDX5d0dcd82c7f0, d_aggr0__tmp_attr0, d_KEY_5d0dcd819160customer__c_city_encoded, d_KEY_5d0dcd819160date__d_year, d_KEY_5d0dcd819160supplier__s_city_encoded);
hipMemcpy(MAT5d0dcd82c7f0customer__c_city_encoded, d_MAT5d0dcd82c7f0customer__c_city_encoded, sizeof(DBI16Type) * COUNT5d0dcd82c7f0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5d0dcd82c7f0supplier__s_city_encoded, d_MAT5d0dcd82c7f0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5d0dcd82c7f0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5d0dcd82c7f0date__d_year, d_MAT5d0dcd82c7f0date__d_year, sizeof(DBI32Type) * COUNT5d0dcd82c7f0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5d0dcd82c7f0aggr0__tmp_attr0, d_MAT5d0dcd82c7f0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5d0dcd82c7f0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5d0dcd82c7f0; i++) { std::cout << "" << customer__c_city_map[MAT5d0dcd82c7f0customer__c_city_encoded[i]];
std::cout << "," << supplier__s_city_map[MAT5d0dcd82c7f0supplier__s_city_encoded[i]];
std::cout << "," << MAT5d0dcd82c7f0date__d_year[i];
std::cout << "," << MAT5d0dcd82c7f0aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_5d0dcd85fc60);
hipFree(d_BUF_IDX_5d0dcd85fc60);
hipFree(d_COUNT5d0dcd85fc60);
hipFree(d_BUF_5d0dcd8686e0);
hipFree(d_BUF_IDX_5d0dcd8686e0);
hipFree(d_COUNT5d0dcd8686e0);
hipFree(d_BUF_5d0dcd860aa0);
hipFree(d_BUF_IDX_5d0dcd860aa0);
hipFree(d_COUNT5d0dcd860aa0);
hipFree(d_KEY_5d0dcd819160customer__c_city_encoded);
hipFree(d_KEY_5d0dcd819160date__d_year);
hipFree(d_KEY_5d0dcd819160supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5d0dcd82c7f0);
hipFree(d_MAT5d0dcd82c7f0aggr0__tmp_attr0);
hipFree(d_MAT5d0dcd82c7f0customer__c_city_encoded);
hipFree(d_MAT5d0dcd82c7f0date__d_year);
hipFree(d_MAT5d0dcd82c7f0supplier__s_city_encoded);
hipFree(d_MAT_IDX5d0dcd82c7f0);
free(MAT5d0dcd82c7f0aggr0__tmp_attr0);
free(MAT5d0dcd82c7f0customer__c_city_encoded);
free(MAT5d0dcd82c7f0date__d_year);
free(MAT5d0dcd82c7f0supplier__s_city_encoded);
}