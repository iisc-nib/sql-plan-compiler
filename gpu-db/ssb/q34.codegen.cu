#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5b6bc68d8c80(uint64_t* COUNT5b6bc68f76e0, DBStringType* customer__c_city, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_city = customer__c_city[tid];
if (!((evaluatePredicate(reg_customer__c_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_customer__c_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5b6bc68f76e0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5b6bc68d8c80(uint64_t* BUF_5b6bc68f76e0, uint64_t* BUF_IDX_5b6bc68f76e0, HASHTABLE_INSERT HT_5b6bc68f76e0, DBStringType* customer__c_city, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_city = customer__c_city[tid];
if (!((evaluatePredicate(reg_customer__c_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_customer__c_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5b6bc68f76e0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5b6bc68f76e0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_5b6bc68f76e0 = atomicAdd((int*)BUF_IDX_5b6bc68f76e0, 1);
HT_5b6bc68f76e0.insert(cuco::pair{KEY_5b6bc68f76e0, buf_idx_5b6bc68f76e0});
BUF_5b6bc68f76e0[buf_idx_5b6bc68f76e0 * 1 + 0] = tid;
}
__global__ void count_5b6bc68d9220(uint64_t* COUNT5b6bc68f23e0, DBStringType* supplier__s_city, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_city = supplier__s_city[tid];
if (!((evaluatePredicate(reg_supplier__s_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_supplier__s_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5b6bc68f23e0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5b6bc68d9220(uint64_t* BUF_5b6bc68f23e0, uint64_t* BUF_IDX_5b6bc68f23e0, HASHTABLE_INSERT HT_5b6bc68f23e0, DBStringType* supplier__s_city, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_city = supplier__s_city[tid];
if (!((evaluatePredicate(reg_supplier__s_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_supplier__s_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5b6bc68f23e0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5b6bc68f23e0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5b6bc68f23e0 = atomicAdd((int*)BUF_IDX_5b6bc68f23e0, 1);
HT_5b6bc68f23e0.insert(cuco::pair{KEY_5b6bc68f23e0, buf_idx_5b6bc68f23e0});
BUF_5b6bc68f23e0[buf_idx_5b6bc68f23e0 * 1 + 0] = tid;
}
__global__ void count_5b6bc69051d0(uint64_t* COUNT5b6bc68f8450, DBStringType* date__d_yearmonth, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_yearmonth = date__d_yearmonth[tid];
if (!(evaluatePredicate(reg_date__d_yearmonth, "Dec1997", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5b6bc68f8450, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5b6bc69051d0(uint64_t* BUF_5b6bc68f8450, uint64_t* BUF_IDX_5b6bc68f8450, HASHTABLE_INSERT HT_5b6bc68f8450, DBI32Type* date__d_datekey, DBStringType* date__d_yearmonth, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_yearmonth = date__d_yearmonth[tid];
if (!(evaluatePredicate(reg_date__d_yearmonth, "Dec1997", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5b6bc68f8450 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5b6bc68f8450 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_5b6bc68f8450 = atomicAdd((int*)BUF_IDX_5b6bc68f8450, 1);
HT_5b6bc68f8450.insert(cuco::pair{KEY_5b6bc68f8450, buf_idx_5b6bc68f8450});
BUF_5b6bc68f8450[buf_idx_5b6bc68f8450 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5b6bc68f6ae0(uint64_t* BUF_5b6bc68f23e0, uint64_t* BUF_5b6bc68f76e0, uint64_t* BUF_5b6bc68f8450, HASHTABLE_INSERT HT_5b6bc68a90f0, HASHTABLE_PROBE HT_5b6bc68f23e0, HASHTABLE_PROBE HT_5b6bc68f76e0, HASHTABLE_PROBE HT_5b6bc68f8450, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5b6bc68f76e0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5b6bc68f76e0 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_5b6bc68f76e0.for_each(KEY_5b6bc68f76e0, [&] __device__ (auto const SLOT_5b6bc68f76e0) {

auto const [slot_first5b6bc68f76e0, slot_second5b6bc68f76e0] = SLOT_5b6bc68f76e0;
if (!(true)) return;
uint64_t KEY_5b6bc68f23e0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5b6bc68f23e0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5b6bc68f23e0.for_each(KEY_5b6bc68f23e0, [&] __device__ (auto const SLOT_5b6bc68f23e0) {

auto const [slot_first5b6bc68f23e0, slot_second5b6bc68f23e0] = SLOT_5b6bc68f23e0;
if (!(true)) return;
uint64_t KEY_5b6bc68f8450 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5b6bc68f8450 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5b6bc68f8450.for_each(KEY_5b6bc68f8450, [&] __device__ (auto const SLOT_5b6bc68f8450) {

auto const [slot_first5b6bc68f8450, slot_second5b6bc68f8450] = SLOT_5b6bc68f8450;
if (!(true)) return;
uint64_t KEY_5b6bc68a90f0 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_5b6bc68f76e0[slot_second5b6bc68f76e0 * 1 + 0]];

KEY_5b6bc68a90f0 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_5b6bc68f23e0[slot_second5b6bc68f23e0 * 1 + 0]];
KEY_5b6bc68a90f0 <<= 16;
KEY_5b6bc68a90f0 |= reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[BUF_5b6bc68f8450[slot_second5b6bc68f8450 * 1 + 0]];
KEY_5b6bc68a90f0 <<= 32;
KEY_5b6bc68a90f0 |= reg_date__d_year;
//Create aggregation hash table
HT_5b6bc68a90f0.insert(cuco::pair{KEY_5b6bc68a90f0, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5b6bc68f6ae0(uint64_t* BUF_5b6bc68f23e0, uint64_t* BUF_5b6bc68f76e0, uint64_t* BUF_5b6bc68f8450, HASHTABLE_FIND HT_5b6bc68a90f0, HASHTABLE_PROBE HT_5b6bc68f23e0, HASHTABLE_PROBE HT_5b6bc68f76e0, HASHTABLE_PROBE HT_5b6bc68f8450, DBI16Type* KEY_5b6bc68a90f0customer__c_city_encoded, DBI32Type* KEY_5b6bc68a90f0date__d_year, DBI16Type* KEY_5b6bc68a90f0supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5b6bc68f76e0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5b6bc68f76e0 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_5b6bc68f76e0.for_each(KEY_5b6bc68f76e0, [&] __device__ (auto const SLOT_5b6bc68f76e0) {
auto const [slot_first5b6bc68f76e0, slot_second5b6bc68f76e0] = SLOT_5b6bc68f76e0;
if (!(true)) return;
uint64_t KEY_5b6bc68f23e0 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5b6bc68f23e0 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5b6bc68f23e0.for_each(KEY_5b6bc68f23e0, [&] __device__ (auto const SLOT_5b6bc68f23e0) {
auto const [slot_first5b6bc68f23e0, slot_second5b6bc68f23e0] = SLOT_5b6bc68f23e0;
if (!(true)) return;
uint64_t KEY_5b6bc68f8450 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5b6bc68f8450 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5b6bc68f8450.for_each(KEY_5b6bc68f8450, [&] __device__ (auto const SLOT_5b6bc68f8450) {
auto const [slot_first5b6bc68f8450, slot_second5b6bc68f8450] = SLOT_5b6bc68f8450;
if (!(true)) return;
uint64_t KEY_5b6bc68a90f0 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_5b6bc68f76e0[slot_second5b6bc68f76e0 * 1 + 0]];

KEY_5b6bc68a90f0 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_5b6bc68f23e0[slot_second5b6bc68f23e0 * 1 + 0]];
KEY_5b6bc68a90f0 <<= 16;
KEY_5b6bc68a90f0 |= reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[BUF_5b6bc68f8450[slot_second5b6bc68f8450 * 1 + 0]];
KEY_5b6bc68a90f0 <<= 32;
KEY_5b6bc68a90f0 |= reg_date__d_year;
//Aggregate in hashtable
auto buf_idx_5b6bc68a90f0 = HT_5b6bc68a90f0.find(KEY_5b6bc68a90f0)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5b6bc68a90f0], reg_lineorder__lo_revenue);
KEY_5b6bc68a90f0customer__c_city_encoded[buf_idx_5b6bc68a90f0] = reg_customer__c_city_encoded;
KEY_5b6bc68a90f0supplier__s_city_encoded[buf_idx_5b6bc68a90f0] = reg_supplier__s_city_encoded;
KEY_5b6bc68a90f0date__d_year[buf_idx_5b6bc68a90f0] = reg_date__d_year;
});
});
});
}
__global__ void count_5b6bc69126d0(size_t COUNT5b6bc68a90f0, uint64_t* COUNT5b6bc68bc240) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5b6bc68a90f0) return;
//Materialize count
atomicAdd((int*)COUNT5b6bc68bc240, 1);
}
__global__ void main_5b6bc69126d0(size_t COUNT5b6bc68a90f0, DBDecimalType* MAT5b6bc68bc240aggr0__tmp_attr0, DBI16Type* MAT5b6bc68bc240customer__c_city_encoded, DBI32Type* MAT5b6bc68bc240date__d_year, DBI16Type* MAT5b6bc68bc240supplier__s_city_encoded, uint64_t* MAT_IDX5b6bc68bc240, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5b6bc68a90f0) return;
//Materialize buffers
auto mat_idx5b6bc68bc240 = atomicAdd((int*)MAT_IDX5b6bc68bc240, 1);
auto reg_customer__c_city_encoded = customer__c_city_encoded[tid];
MAT5b6bc68bc240customer__c_city_encoded[mat_idx5b6bc68bc240] = reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[tid];
MAT5b6bc68bc240supplier__s_city_encoded[mat_idx5b6bc68bc240] = reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[tid];
MAT5b6bc68bc240date__d_year[mat_idx5b6bc68bc240] = reg_date__d_year;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5b6bc68bc240aggr0__tmp_attr0[mat_idx5b6bc68bc240] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT5b6bc68f76e0;
hipMalloc(&d_COUNT5b6bc68f76e0, sizeof(uint64_t));
hipMemset(d_COUNT5b6bc68f76e0, 0, sizeof(uint64_t));
count_5b6bc68d8c80<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT5b6bc68f76e0, d_customer__c_city, customer_size);
uint64_t COUNT5b6bc68f76e0;
hipMemcpy(&COUNT5b6bc68f76e0, d_COUNT5b6bc68f76e0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b6bc68f76e0;
hipMalloc(&d_BUF_IDX_5b6bc68f76e0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b6bc68f76e0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b6bc68f76e0;
hipMalloc(&d_BUF_5b6bc68f76e0, sizeof(uint64_t) * COUNT5b6bc68f76e0 * 1);
auto d_HT_5b6bc68f76e0 = cuco::experimental::static_multimap{ (int)COUNT5b6bc68f76e0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b6bc68d8c80<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5b6bc68f76e0, d_BUF_IDX_5b6bc68f76e0, d_HT_5b6bc68f76e0.ref(cuco::insert), d_customer__c_city, d_customer__c_custkey, customer_size);
//Materialize count
uint64_t* d_COUNT5b6bc68f23e0;
hipMalloc(&d_COUNT5b6bc68f23e0, sizeof(uint64_t));
hipMemset(d_COUNT5b6bc68f23e0, 0, sizeof(uint64_t));
count_5b6bc68d9220<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT5b6bc68f23e0, d_supplier__s_city, supplier_size);
uint64_t COUNT5b6bc68f23e0;
hipMemcpy(&COUNT5b6bc68f23e0, d_COUNT5b6bc68f23e0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b6bc68f23e0;
hipMalloc(&d_BUF_IDX_5b6bc68f23e0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b6bc68f23e0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b6bc68f23e0;
hipMalloc(&d_BUF_5b6bc68f23e0, sizeof(uint64_t) * COUNT5b6bc68f23e0 * 1);
auto d_HT_5b6bc68f23e0 = cuco::experimental::static_multimap{ (int)COUNT5b6bc68f23e0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b6bc68d9220<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_5b6bc68f23e0, d_BUF_IDX_5b6bc68f23e0, d_HT_5b6bc68f23e0.ref(cuco::insert), d_supplier__s_city, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5b6bc68f8450;
hipMalloc(&d_COUNT5b6bc68f8450, sizeof(uint64_t));
hipMemset(d_COUNT5b6bc68f8450, 0, sizeof(uint64_t));
count_5b6bc69051d0<<<std::ceil((float)date_size/32.), 32>>>(d_COUNT5b6bc68f8450, d_date__d_yearmonth, date_size);
uint64_t COUNT5b6bc68f8450;
hipMemcpy(&COUNT5b6bc68f8450, d_COUNT5b6bc68f8450, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b6bc68f8450;
hipMalloc(&d_BUF_IDX_5b6bc68f8450, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b6bc68f8450, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b6bc68f8450;
hipMalloc(&d_BUF_5b6bc68f8450, sizeof(uint64_t) * COUNT5b6bc68f8450 * 1);
auto d_HT_5b6bc68f8450 = cuco::experimental::static_multimap{ (int)COUNT5b6bc68f8450*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b6bc69051d0<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_5b6bc68f8450, d_BUF_IDX_5b6bc68f8450, d_HT_5b6bc68f8450.ref(cuco::insert), d_date__d_datekey, d_date__d_yearmonth, date_size);
//Create aggregation hash table
auto d_HT_5b6bc68a90f0 = cuco::static_map{ (int)3*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5b6bc68f6ae0<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_5b6bc68f23e0, d_BUF_5b6bc68f76e0, d_BUF_5b6bc68f8450, d_HT_5b6bc68a90f0.ref(cuco::insert), d_HT_5b6bc68f23e0.ref(cuco::for_each), d_HT_5b6bc68f76e0.ref(cuco::for_each), d_HT_5b6bc68f8450.ref(cuco::for_each), d_customer__c_city_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
size_t COUNT5b6bc68a90f0 = d_HT_5b6bc68a90f0.size();
thrust::device_vector<int64_t> keys_5b6bc68a90f0(COUNT5b6bc68a90f0), vals_5b6bc68a90f0(COUNT5b6bc68a90f0);
d_HT_5b6bc68a90f0.retrieve_all(keys_5b6bc68a90f0.begin(), vals_5b6bc68a90f0.begin());
d_HT_5b6bc68a90f0.clear();
int64_t* raw_keys5b6bc68a90f0 = thrust::raw_pointer_cast(keys_5b6bc68a90f0.data());
insertKeys<<<std::ceil((float)COUNT5b6bc68a90f0/32.), 32>>>(raw_keys5b6bc68a90f0, d_HT_5b6bc68a90f0.ref(cuco::insert), COUNT5b6bc68a90f0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5b6bc68a90f0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5b6bc68a90f0);
DBI16Type* d_KEY_5b6bc68a90f0customer__c_city_encoded;
hipMalloc(&d_KEY_5b6bc68a90f0customer__c_city_encoded, sizeof(DBI16Type) * COUNT5b6bc68a90f0);
hipMemset(d_KEY_5b6bc68a90f0customer__c_city_encoded, 0, sizeof(DBI16Type) * COUNT5b6bc68a90f0);
DBI16Type* d_KEY_5b6bc68a90f0supplier__s_city_encoded;
hipMalloc(&d_KEY_5b6bc68a90f0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5b6bc68a90f0);
hipMemset(d_KEY_5b6bc68a90f0supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT5b6bc68a90f0);
DBI32Type* d_KEY_5b6bc68a90f0date__d_year;
hipMalloc(&d_KEY_5b6bc68a90f0date__d_year, sizeof(DBI32Type) * COUNT5b6bc68a90f0);
hipMemset(d_KEY_5b6bc68a90f0date__d_year, 0, sizeof(DBI32Type) * COUNT5b6bc68a90f0);
main_5b6bc68f6ae0<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_5b6bc68f23e0, d_BUF_5b6bc68f76e0, d_BUF_5b6bc68f8450, d_HT_5b6bc68a90f0.ref(cuco::find), d_HT_5b6bc68f23e0.ref(cuco::for_each), d_HT_5b6bc68f76e0.ref(cuco::for_each), d_HT_5b6bc68f8450.ref(cuco::for_each), d_KEY_5b6bc68a90f0customer__c_city_encoded, d_KEY_5b6bc68a90f0date__d_year, d_KEY_5b6bc68a90f0supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_city_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
//Materialize count
uint64_t* d_COUNT5b6bc68bc240;
hipMalloc(&d_COUNT5b6bc68bc240, sizeof(uint64_t));
hipMemset(d_COUNT5b6bc68bc240, 0, sizeof(uint64_t));
count_5b6bc69126d0<<<std::ceil((float)COUNT5b6bc68a90f0/32.), 32>>>(COUNT5b6bc68a90f0, d_COUNT5b6bc68bc240);
uint64_t COUNT5b6bc68bc240;
hipMemcpy(&COUNT5b6bc68bc240, d_COUNT5b6bc68bc240, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5b6bc68bc240;
hipMalloc(&d_MAT_IDX5b6bc68bc240, sizeof(uint64_t));
hipMemset(d_MAT_IDX5b6bc68bc240, 0, sizeof(uint64_t));
auto MAT5b6bc68bc240customer__c_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5b6bc68bc240);
DBI16Type* d_MAT5b6bc68bc240customer__c_city_encoded;
hipMalloc(&d_MAT5b6bc68bc240customer__c_city_encoded, sizeof(DBI16Type) * COUNT5b6bc68bc240);
auto MAT5b6bc68bc240supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5b6bc68bc240);
DBI16Type* d_MAT5b6bc68bc240supplier__s_city_encoded;
hipMalloc(&d_MAT5b6bc68bc240supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5b6bc68bc240);
auto MAT5b6bc68bc240date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5b6bc68bc240);
DBI32Type* d_MAT5b6bc68bc240date__d_year;
hipMalloc(&d_MAT5b6bc68bc240date__d_year, sizeof(DBI32Type) * COUNT5b6bc68bc240);
auto MAT5b6bc68bc240aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5b6bc68bc240);
DBDecimalType* d_MAT5b6bc68bc240aggr0__tmp_attr0;
hipMalloc(&d_MAT5b6bc68bc240aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5b6bc68bc240);
main_5b6bc69126d0<<<std::ceil((float)COUNT5b6bc68a90f0/32.), 32>>>(COUNT5b6bc68a90f0, d_MAT5b6bc68bc240aggr0__tmp_attr0, d_MAT5b6bc68bc240customer__c_city_encoded, d_MAT5b6bc68bc240date__d_year, d_MAT5b6bc68bc240supplier__s_city_encoded, d_MAT_IDX5b6bc68bc240, d_aggr0__tmp_attr0, d_KEY_5b6bc68a90f0customer__c_city_encoded, d_KEY_5b6bc68a90f0date__d_year, d_KEY_5b6bc68a90f0supplier__s_city_encoded);
hipMemcpy(MAT5b6bc68bc240customer__c_city_encoded, d_MAT5b6bc68bc240customer__c_city_encoded, sizeof(DBI16Type) * COUNT5b6bc68bc240, hipMemcpyDeviceToHost);
hipMemcpy(MAT5b6bc68bc240supplier__s_city_encoded, d_MAT5b6bc68bc240supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5b6bc68bc240, hipMemcpyDeviceToHost);
hipMemcpy(MAT5b6bc68bc240date__d_year, d_MAT5b6bc68bc240date__d_year, sizeof(DBI32Type) * COUNT5b6bc68bc240, hipMemcpyDeviceToHost);
hipMemcpy(MAT5b6bc68bc240aggr0__tmp_attr0, d_MAT5b6bc68bc240aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5b6bc68bc240, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5b6bc68bc240; i++) { std::cout << customer__c_city_map[MAT5b6bc68bc240customer__c_city_encoded[i]] << "\t";
std::cout << supplier__s_city_map[MAT5b6bc68bc240supplier__s_city_encoded[i]] << "\t";
std::cout << MAT5b6bc68bc240date__d_year[i] << "\t";
std::cout << MAT5b6bc68bc240aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_5b6bc68f76e0);
hipFree(d_BUF_IDX_5b6bc68f76e0);
hipFree(d_COUNT5b6bc68f76e0);
hipFree(d_BUF_5b6bc68f23e0);
hipFree(d_BUF_IDX_5b6bc68f23e0);
hipFree(d_COUNT5b6bc68f23e0);
hipFree(d_BUF_5b6bc68f8450);
hipFree(d_BUF_IDX_5b6bc68f8450);
hipFree(d_COUNT5b6bc68f8450);
hipFree(d_KEY_5b6bc68a90f0customer__c_city_encoded);
hipFree(d_KEY_5b6bc68a90f0date__d_year);
hipFree(d_KEY_5b6bc68a90f0supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5b6bc68bc240);
hipFree(d_MAT5b6bc68bc240aggr0__tmp_attr0);
hipFree(d_MAT5b6bc68bc240customer__c_city_encoded);
hipFree(d_MAT5b6bc68bc240date__d_year);
hipFree(d_MAT5b6bc68bc240supplier__s_city_encoded);
hipFree(d_MAT_IDX5b6bc68bc240);
free(MAT5b6bc68bc240aggr0__tmp_attr0);
free(MAT5b6bc68bc240customer__c_city_encoded);
free(MAT5b6bc68bc240date__d_year);
free(MAT5b6bc68bc240supplier__s_city_encoded);
}