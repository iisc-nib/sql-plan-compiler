#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_6487e32e9610(uint64_t* COUNT6487e32ffd80, DBStringType* customer__c_city, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_city = customer__c_city[tid];
if (!((evaluatePredicate(reg_customer__c_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_customer__c_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT6487e32ffd80, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_6487e32e9610(uint64_t* BUF_6487e32ffd80, uint64_t* BUF_IDX_6487e32ffd80, HASHTABLE_INSERT HT_6487e32ffd80, DBStringType* customer__c_city, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_city = customer__c_city[tid];
if (!((evaluatePredicate(reg_customer__c_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_customer__c_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_6487e32ffd80 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_6487e32ffd80 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_6487e32ffd80 = atomicAdd((int*)BUF_IDX_6487e32ffd80, 1);
HT_6487e32ffd80.insert(cuco::pair{KEY_6487e32ffd80, buf_idx_6487e32ffd80});
BUF_6487e32ffd80[buf_idx_6487e32ffd80 * 1 + 0] = tid;
}
__global__ void count_6487e32e9bb0(uint64_t* COUNT6487e3303f30, DBStringType* supplier__s_city, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_city = supplier__s_city[tid];
if (!((evaluatePredicate(reg_supplier__s_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_supplier__s_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT6487e3303f30, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_6487e32e9bb0(uint64_t* BUF_6487e3303f30, uint64_t* BUF_IDX_6487e3303f30, HASHTABLE_INSERT HT_6487e3303f30, DBStringType* supplier__s_city, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_city = supplier__s_city[tid];
if (!((evaluatePredicate(reg_supplier__s_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_supplier__s_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_6487e3303f30 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_6487e3303f30 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_6487e3303f30 = atomicAdd((int*)BUF_IDX_6487e3303f30, 1);
HT_6487e3303f30.insert(cuco::pair{KEY_6487e3303f30, buf_idx_6487e3303f30});
BUF_6487e3303f30[buf_idx_6487e3303f30 * 1 + 0] = tid;
}
__global__ void count_6487e3315010(uint64_t* COUNT6487e32ffe40, DBStringType* date__d_yearmonth, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_yearmonth = date__d_yearmonth[tid];
if (!(evaluatePredicate(reg_date__d_yearmonth, "Dec1997", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT6487e32ffe40, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_6487e3315010(uint64_t* BUF_6487e32ffe40, uint64_t* BUF_IDX_6487e32ffe40, HASHTABLE_INSERT HT_6487e32ffe40, DBI32Type* date__d_datekey, DBStringType* date__d_yearmonth, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_yearmonth = date__d_yearmonth[tid];
if (!(evaluatePredicate(reg_date__d_yearmonth, "Dec1997", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_6487e32ffe40 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_6487e32ffe40 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_6487e32ffe40 = atomicAdd((int*)BUF_IDX_6487e32ffe40, 1);
HT_6487e32ffe40.insert(cuco::pair{KEY_6487e32ffe40, buf_idx_6487e32ffe40});
BUF_6487e32ffe40[buf_idx_6487e32ffe40 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_6487e33029e0(uint64_t* BUF_6487e32ffd80, uint64_t* BUF_6487e32ffe40, uint64_t* BUF_6487e3303f30, HASHTABLE_INSERT HT_6487e32b9de0, HASHTABLE_PROBE HT_6487e32ffd80, HASHTABLE_PROBE HT_6487e32ffe40, HASHTABLE_PROBE HT_6487e3303f30, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_6487e32ffd80 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_6487e32ffd80 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_6487e32ffd80.for_each(KEY_6487e32ffd80, [&] __device__ (auto const SLOT_6487e32ffd80) {

auto const [slot_first6487e32ffd80, slot_second6487e32ffd80] = SLOT_6487e32ffd80;
if (!(true)) return;
uint64_t KEY_6487e3303f30 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_6487e3303f30 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_6487e3303f30.for_each(KEY_6487e3303f30, [&] __device__ (auto const SLOT_6487e3303f30) {

auto const [slot_first6487e3303f30, slot_second6487e3303f30] = SLOT_6487e3303f30;
if (!(true)) return;
uint64_t KEY_6487e32ffe40 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_6487e32ffe40 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_6487e32ffe40.for_each(KEY_6487e32ffe40, [&] __device__ (auto const SLOT_6487e32ffe40) {

auto const [slot_first6487e32ffe40, slot_second6487e32ffe40] = SLOT_6487e32ffe40;
if (!(true)) return;
uint64_t KEY_6487e32b9de0 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_6487e32ffd80[slot_second6487e32ffd80 * 1 + 0]];

KEY_6487e32b9de0 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_6487e3303f30[slot_second6487e3303f30 * 1 + 0]];
KEY_6487e32b9de0 <<= 16;
KEY_6487e32b9de0 |= reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[BUF_6487e32ffe40[slot_second6487e32ffe40 * 1 + 0]];
KEY_6487e32b9de0 <<= 32;
KEY_6487e32b9de0 |= reg_date__d_year;
//Create aggregation hash table
HT_6487e32b9de0.insert(cuco::pair{KEY_6487e32b9de0, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_6487e33029e0(uint64_t* BUF_6487e32ffd80, uint64_t* BUF_6487e32ffe40, uint64_t* BUF_6487e3303f30, HASHTABLE_FIND HT_6487e32b9de0, HASHTABLE_PROBE HT_6487e32ffd80, HASHTABLE_PROBE HT_6487e32ffe40, HASHTABLE_PROBE HT_6487e3303f30, DBI16Type* KEY_6487e32b9de0customer__c_city_encoded, DBI32Type* KEY_6487e32b9de0date__d_year, DBI16Type* KEY_6487e32b9de0supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_6487e32ffd80 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_6487e32ffd80 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_6487e32ffd80.for_each(KEY_6487e32ffd80, [&] __device__ (auto const SLOT_6487e32ffd80) {
auto const [slot_first6487e32ffd80, slot_second6487e32ffd80] = SLOT_6487e32ffd80;
if (!(true)) return;
uint64_t KEY_6487e3303f30 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_6487e3303f30 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_6487e3303f30.for_each(KEY_6487e3303f30, [&] __device__ (auto const SLOT_6487e3303f30) {
auto const [slot_first6487e3303f30, slot_second6487e3303f30] = SLOT_6487e3303f30;
if (!(true)) return;
uint64_t KEY_6487e32ffe40 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_6487e32ffe40 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_6487e32ffe40.for_each(KEY_6487e32ffe40, [&] __device__ (auto const SLOT_6487e32ffe40) {
auto const [slot_first6487e32ffe40, slot_second6487e32ffe40] = SLOT_6487e32ffe40;
if (!(true)) return;
uint64_t KEY_6487e32b9de0 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_6487e32ffd80[slot_second6487e32ffd80 * 1 + 0]];

KEY_6487e32b9de0 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_6487e3303f30[slot_second6487e3303f30 * 1 + 0]];
KEY_6487e32b9de0 <<= 16;
KEY_6487e32b9de0 |= reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[BUF_6487e32ffe40[slot_second6487e32ffe40 * 1 + 0]];
KEY_6487e32b9de0 <<= 32;
KEY_6487e32b9de0 |= reg_date__d_year;
//Aggregate in hashtable
auto buf_idx_6487e32b9de0 = HT_6487e32b9de0.find(KEY_6487e32b9de0)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_6487e32b9de0], reg_lineorder__lo_revenue);
KEY_6487e32b9de0customer__c_city_encoded[buf_idx_6487e32b9de0] = reg_customer__c_city_encoded;
KEY_6487e32b9de0supplier__s_city_encoded[buf_idx_6487e32b9de0] = reg_supplier__s_city_encoded;
KEY_6487e32b9de0date__d_year[buf_idx_6487e32b9de0] = reg_date__d_year;
});
});
});
}
__global__ void count_6487e3322050(size_t COUNT6487e32b9de0, uint64_t* COUNT6487e32ccf90) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT6487e32b9de0) return;
//Materialize count
atomicAdd((int*)COUNT6487e32ccf90, 1);
}
__global__ void main_6487e3322050(size_t COUNT6487e32b9de0, DBDecimalType* MAT6487e32ccf90aggr0__tmp_attr0, DBI16Type* MAT6487e32ccf90customer__c_city_encoded, DBI32Type* MAT6487e32ccf90date__d_year, DBI16Type* MAT6487e32ccf90supplier__s_city_encoded, uint64_t* MAT_IDX6487e32ccf90, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT6487e32b9de0) return;
//Materialize buffers
auto mat_idx6487e32ccf90 = atomicAdd((int*)MAT_IDX6487e32ccf90, 1);
auto reg_customer__c_city_encoded = customer__c_city_encoded[tid];
MAT6487e32ccf90customer__c_city_encoded[mat_idx6487e32ccf90] = reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[tid];
MAT6487e32ccf90supplier__s_city_encoded[mat_idx6487e32ccf90] = reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[tid];
MAT6487e32ccf90date__d_year[mat_idx6487e32ccf90] = reg_date__d_year;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT6487e32ccf90aggr0__tmp_attr0[mat_idx6487e32ccf90] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map) {
//Materialize count
uint64_t* d_COUNT6487e32ffd80;
hipMalloc(&d_COUNT6487e32ffd80, sizeof(uint64_t));
hipMemset(d_COUNT6487e32ffd80, 0, sizeof(uint64_t));
count_6487e32e9610<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT6487e32ffd80, d_customer__c_city, customer_size);
uint64_t COUNT6487e32ffd80;
hipMemcpy(&COUNT6487e32ffd80, d_COUNT6487e32ffd80, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_6487e32ffd80;
hipMalloc(&d_BUF_IDX_6487e32ffd80, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6487e32ffd80, 0, sizeof(uint64_t));
uint64_t* d_BUF_6487e32ffd80;
hipMalloc(&d_BUF_6487e32ffd80, sizeof(uint64_t) * COUNT6487e32ffd80 * 1);
auto d_HT_6487e32ffd80 = cuco::experimental::static_multimap{ (int)COUNT6487e32ffd80*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_6487e32e9610<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_6487e32ffd80, d_BUF_IDX_6487e32ffd80, d_HT_6487e32ffd80.ref(cuco::insert), d_customer__c_city, d_customer__c_custkey, customer_size);
//Materialize count
uint64_t* d_COUNT6487e3303f30;
hipMalloc(&d_COUNT6487e3303f30, sizeof(uint64_t));
hipMemset(d_COUNT6487e3303f30, 0, sizeof(uint64_t));
count_6487e32e9bb0<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT6487e3303f30, d_supplier__s_city, supplier_size);
uint64_t COUNT6487e3303f30;
hipMemcpy(&COUNT6487e3303f30, d_COUNT6487e3303f30, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_6487e3303f30;
hipMalloc(&d_BUF_IDX_6487e3303f30, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6487e3303f30, 0, sizeof(uint64_t));
uint64_t* d_BUF_6487e3303f30;
hipMalloc(&d_BUF_6487e3303f30, sizeof(uint64_t) * COUNT6487e3303f30 * 1);
auto d_HT_6487e3303f30 = cuco::experimental::static_multimap{ (int)COUNT6487e3303f30*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_6487e32e9bb0<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_6487e3303f30, d_BUF_IDX_6487e3303f30, d_HT_6487e3303f30.ref(cuco::insert), d_supplier__s_city, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT6487e32ffe40;
hipMalloc(&d_COUNT6487e32ffe40, sizeof(uint64_t));
hipMemset(d_COUNT6487e32ffe40, 0, sizeof(uint64_t));
count_6487e3315010<<<std::ceil((float)date_size/32.), 32>>>(d_COUNT6487e32ffe40, d_date__d_yearmonth, date_size);
uint64_t COUNT6487e32ffe40;
hipMemcpy(&COUNT6487e32ffe40, d_COUNT6487e32ffe40, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_6487e32ffe40;
hipMalloc(&d_BUF_IDX_6487e32ffe40, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6487e32ffe40, 0, sizeof(uint64_t));
uint64_t* d_BUF_6487e32ffe40;
hipMalloc(&d_BUF_6487e32ffe40, sizeof(uint64_t) * COUNT6487e32ffe40 * 1);
auto d_HT_6487e32ffe40 = cuco::experimental::static_multimap{ (int)COUNT6487e32ffe40*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_6487e3315010<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_6487e32ffe40, d_BUF_IDX_6487e32ffe40, d_HT_6487e32ffe40.ref(cuco::insert), d_date__d_datekey, d_date__d_yearmonth, date_size);
//Create aggregation hash table
auto d_HT_6487e32b9de0 = cuco::static_map{ (int)3*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_6487e33029e0<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_6487e32ffd80, d_BUF_6487e32ffe40, d_BUF_6487e3303f30, d_HT_6487e32b9de0.ref(cuco::insert), d_HT_6487e32ffd80.ref(cuco::for_each), d_HT_6487e32ffe40.ref(cuco::for_each), d_HT_6487e3303f30.ref(cuco::for_each), d_customer__c_city_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
size_t COUNT6487e32b9de0 = d_HT_6487e32b9de0.size();
thrust::device_vector<int64_t> keys_6487e32b9de0(COUNT6487e32b9de0), vals_6487e32b9de0(COUNT6487e32b9de0);
d_HT_6487e32b9de0.retrieve_all(keys_6487e32b9de0.begin(), vals_6487e32b9de0.begin());
d_HT_6487e32b9de0.clear();
int64_t* raw_keys6487e32b9de0 = thrust::raw_pointer_cast(keys_6487e32b9de0.data());
insertKeys<<<std::ceil((float)COUNT6487e32b9de0/32.), 32>>>(raw_keys6487e32b9de0, d_HT_6487e32b9de0.ref(cuco::insert), COUNT6487e32b9de0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6487e32b9de0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT6487e32b9de0);
DBI16Type* d_KEY_6487e32b9de0customer__c_city_encoded;
hipMalloc(&d_KEY_6487e32b9de0customer__c_city_encoded, sizeof(DBI16Type) * COUNT6487e32b9de0);
hipMemset(d_KEY_6487e32b9de0customer__c_city_encoded, 0, sizeof(DBI16Type) * COUNT6487e32b9de0);
DBI16Type* d_KEY_6487e32b9de0supplier__s_city_encoded;
hipMalloc(&d_KEY_6487e32b9de0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT6487e32b9de0);
hipMemset(d_KEY_6487e32b9de0supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT6487e32b9de0);
DBI32Type* d_KEY_6487e32b9de0date__d_year;
hipMalloc(&d_KEY_6487e32b9de0date__d_year, sizeof(DBI32Type) * COUNT6487e32b9de0);
hipMemset(d_KEY_6487e32b9de0date__d_year, 0, sizeof(DBI32Type) * COUNT6487e32b9de0);
main_6487e33029e0<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_6487e32ffd80, d_BUF_6487e32ffe40, d_BUF_6487e3303f30, d_HT_6487e32b9de0.ref(cuco::find), d_HT_6487e32ffd80.ref(cuco::for_each), d_HT_6487e32ffe40.ref(cuco::for_each), d_HT_6487e3303f30.ref(cuco::for_each), d_KEY_6487e32b9de0customer__c_city_encoded, d_KEY_6487e32b9de0date__d_year, d_KEY_6487e32b9de0supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_city_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
//Materialize count
uint64_t* d_COUNT6487e32ccf90;
hipMalloc(&d_COUNT6487e32ccf90, sizeof(uint64_t));
hipMemset(d_COUNT6487e32ccf90, 0, sizeof(uint64_t));
count_6487e3322050<<<std::ceil((float)COUNT6487e32b9de0/32.), 32>>>(COUNT6487e32b9de0, d_COUNT6487e32ccf90);
uint64_t COUNT6487e32ccf90;
hipMemcpy(&COUNT6487e32ccf90, d_COUNT6487e32ccf90, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX6487e32ccf90;
hipMalloc(&d_MAT_IDX6487e32ccf90, sizeof(uint64_t));
hipMemset(d_MAT_IDX6487e32ccf90, 0, sizeof(uint64_t));
auto MAT6487e32ccf90customer__c_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT6487e32ccf90);
DBI16Type* d_MAT6487e32ccf90customer__c_city_encoded;
hipMalloc(&d_MAT6487e32ccf90customer__c_city_encoded, sizeof(DBI16Type) * COUNT6487e32ccf90);
auto MAT6487e32ccf90supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT6487e32ccf90);
DBI16Type* d_MAT6487e32ccf90supplier__s_city_encoded;
hipMalloc(&d_MAT6487e32ccf90supplier__s_city_encoded, sizeof(DBI16Type) * COUNT6487e32ccf90);
auto MAT6487e32ccf90date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT6487e32ccf90);
DBI32Type* d_MAT6487e32ccf90date__d_year;
hipMalloc(&d_MAT6487e32ccf90date__d_year, sizeof(DBI32Type) * COUNT6487e32ccf90);
auto MAT6487e32ccf90aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT6487e32ccf90);
DBDecimalType* d_MAT6487e32ccf90aggr0__tmp_attr0;
hipMalloc(&d_MAT6487e32ccf90aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6487e32ccf90);
main_6487e3322050<<<std::ceil((float)COUNT6487e32b9de0/32.), 32>>>(COUNT6487e32b9de0, d_MAT6487e32ccf90aggr0__tmp_attr0, d_MAT6487e32ccf90customer__c_city_encoded, d_MAT6487e32ccf90date__d_year, d_MAT6487e32ccf90supplier__s_city_encoded, d_MAT_IDX6487e32ccf90, d_aggr0__tmp_attr0, d_KEY_6487e32b9de0customer__c_city_encoded, d_KEY_6487e32b9de0date__d_year, d_KEY_6487e32b9de0supplier__s_city_encoded);
hipMemcpy(MAT6487e32ccf90customer__c_city_encoded, d_MAT6487e32ccf90customer__c_city_encoded, sizeof(DBI16Type) * COUNT6487e32ccf90, hipMemcpyDeviceToHost);
hipMemcpy(MAT6487e32ccf90supplier__s_city_encoded, d_MAT6487e32ccf90supplier__s_city_encoded, sizeof(DBI16Type) * COUNT6487e32ccf90, hipMemcpyDeviceToHost);
hipMemcpy(MAT6487e32ccf90date__d_year, d_MAT6487e32ccf90date__d_year, sizeof(DBI32Type) * COUNT6487e32ccf90, hipMemcpyDeviceToHost);
hipMemcpy(MAT6487e32ccf90aggr0__tmp_attr0, d_MAT6487e32ccf90aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6487e32ccf90, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT6487e32ccf90; i++) { std::cout << customer__c_city_map[MAT6487e32ccf90customer__c_city_encoded[i]] << "\t";
std::cout << supplier__s_city_map[MAT6487e32ccf90supplier__s_city_encoded[i]] << "\t";
std::cout << MAT6487e32ccf90date__d_year[i] << "\t";
std::cout << MAT6487e32ccf90aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_6487e32ffd80);
hipFree(d_BUF_IDX_6487e32ffd80);
hipFree(d_COUNT6487e32ffd80);
hipFree(d_BUF_6487e3303f30);
hipFree(d_BUF_IDX_6487e3303f30);
hipFree(d_COUNT6487e3303f30);
hipFree(d_BUF_6487e32ffe40);
hipFree(d_BUF_IDX_6487e32ffe40);
hipFree(d_COUNT6487e32ffe40);
hipFree(d_KEY_6487e32b9de0customer__c_city_encoded);
hipFree(d_KEY_6487e32b9de0date__d_year);
hipFree(d_KEY_6487e32b9de0supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT6487e32ccf90);
hipFree(d_MAT6487e32ccf90aggr0__tmp_attr0);
hipFree(d_MAT6487e32ccf90customer__c_city_encoded);
hipFree(d_MAT6487e32ccf90date__d_year);
hipFree(d_MAT6487e32ccf90supplier__s_city_encoded);
hipFree(d_MAT_IDX6487e32ccf90);
free(MAT6487e32ccf90aggr0__tmp_attr0);
free(MAT6487e32ccf90customer__c_city_encoded);
free(MAT6487e32ccf90date__d_year);
free(MAT6487e32ccf90supplier__s_city_encoded);
}