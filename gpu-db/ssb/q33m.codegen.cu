#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_63fdb23d1120(uint64_t* COUNT63fdb23f0c60, DBStringType* customer__c_city, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_city = customer__c_city[tid];
if (!((evaluatePredicate(reg_customer__c_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_customer__c_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT63fdb23f0c60, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_63fdb23d1120(uint64_t* BUF_63fdb23f0c60, uint64_t* BUF_IDX_63fdb23f0c60, HASHTABLE_INSERT HT_63fdb23f0c60, DBStringType* customer__c_city, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_city = customer__c_city[tid];
if (!((evaluatePredicate(reg_customer__c_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_customer__c_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_63fdb23f0c60 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_63fdb23f0c60 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_63fdb23f0c60 = atomicAdd((int*)BUF_IDX_63fdb23f0c60, 1);
HT_63fdb23f0c60.insert(cuco::pair{KEY_63fdb23f0c60, buf_idx_63fdb23f0c60});
BUF_63fdb23f0c60[buf_idx_63fdb23f0c60 * 1 + 0] = tid;
}
__global__ void count_63fdb23d16c0(uint64_t* COUNT63fdb23ee570, DBStringType* supplier__s_city, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_city = supplier__s_city[tid];
if (!((evaluatePredicate(reg_supplier__s_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_supplier__s_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT63fdb23ee570, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_63fdb23d16c0(uint64_t* BUF_63fdb23ee570, uint64_t* BUF_IDX_63fdb23ee570, HASHTABLE_INSERT HT_63fdb23ee570, DBStringType* supplier__s_city, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_city = supplier__s_city[tid];
if (!((evaluatePredicate(reg_supplier__s_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_supplier__s_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_63fdb23ee570 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_63fdb23ee570 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_63fdb23ee570 = atomicAdd((int*)BUF_IDX_63fdb23ee570, 1);
HT_63fdb23ee570.insert(cuco::pair{KEY_63fdb23ee570, buf_idx_63fdb23ee570});
BUF_63fdb23ee570[buf_idx_63fdb23ee570 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_63fdb23eb550(uint64_t* BUF_63fdb23ee570, uint64_t* BUF_63fdb23f0c60, uint64_t* COUNT63fdb23e9790, HASHTABLE_PROBE HT_63fdb23ee570, HASHTABLE_PROBE HT_63fdb23f0c60, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_63fdb23f0c60 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_63fdb23f0c60 |= reg_lineorder__lo_custkey;
//Probe Hash table
auto SLOT_63fdb23f0c60 = HT_63fdb23f0c60.find(KEY_63fdb23f0c60);
if (SLOT_63fdb23f0c60 == HT_63fdb23f0c60.end()) return;
if (!(true)) return;
uint64_t KEY_63fdb23ee570 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_63fdb23ee570 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_63fdb23ee570 = HT_63fdb23ee570.find(KEY_63fdb23ee570);
if (SLOT_63fdb23ee570 == HT_63fdb23ee570.end()) return;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT63fdb23e9790, 1);
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_63fdb23eb550(uint64_t* BUF_63fdb23e9790, uint64_t* BUF_63fdb23ee570, uint64_t* BUF_63fdb23f0c60, uint64_t* BUF_IDX_63fdb23e9790, HASHTABLE_INSERT HT_63fdb23e9790, HASHTABLE_PROBE HT_63fdb23ee570, HASHTABLE_PROBE HT_63fdb23f0c60, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_63fdb23f0c60 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_63fdb23f0c60 |= reg_lineorder__lo_custkey;
//Probe Hash table
auto SLOT_63fdb23f0c60 = HT_63fdb23f0c60.find(KEY_63fdb23f0c60);
if (SLOT_63fdb23f0c60 == HT_63fdb23f0c60.end()) return;
if (!(true)) return;
uint64_t KEY_63fdb23ee570 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_63fdb23ee570 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_63fdb23ee570 = HT_63fdb23ee570.find(KEY_63fdb23ee570);
if (SLOT_63fdb23ee570 == HT_63fdb23ee570.end()) return;
if (!(true)) return;
uint64_t KEY_63fdb23e9790 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_63fdb23e9790 |= reg_lineorder__lo_orderdate;
// Insert hash table kernel;
auto buf_idx_63fdb23e9790 = atomicAdd((int*)BUF_IDX_63fdb23e9790, 1);
HT_63fdb23e9790.insert(cuco::pair{KEY_63fdb23e9790, buf_idx_63fdb23e9790});
BUF_63fdb23e9790[buf_idx_63fdb23e9790 * 3 + 0] = BUF_63fdb23f0c60[SLOT_63fdb23f0c60->second * 1 + 0];
BUF_63fdb23e9790[buf_idx_63fdb23e9790 * 3 + 1] = tid;
BUF_63fdb23e9790[buf_idx_63fdb23e9790 * 3 + 2] = BUF_63fdb23ee570[SLOT_63fdb23ee570->second * 1 + 0];
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_63fdb23fd4f0(uint64_t* BUF_63fdb23e9790, HASHTABLE_INSERT HT_63fdb23a14e0, HASHTABLE_PROBE HT_63fdb23e9790, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_63fdb23e9790 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_63fdb23e9790 |= reg_date__d_datekey;
//Probe Hash table
auto SLOT_63fdb23e9790 = HT_63fdb23e9790.find(KEY_63fdb23e9790);
if (SLOT_63fdb23e9790 == HT_63fdb23e9790.end()) return;
if (!(true)) return;
uint64_t KEY_63fdb23a14e0 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_63fdb23e9790[SLOT_63fdb23e9790->second * 3 + 0]];

KEY_63fdb23a14e0 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_63fdb23e9790[SLOT_63fdb23e9790->second * 3 + 2]];
KEY_63fdb23a14e0 <<= 16;
KEY_63fdb23a14e0 |= reg_supplier__s_city_encoded;
KEY_63fdb23a14e0 <<= 32;
KEY_63fdb23a14e0 |= reg_date__d_year;
//Create aggregation hash table
HT_63fdb23a14e0.insert(cuco::pair{KEY_63fdb23a14e0, 1});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_63fdb23fd4f0(uint64_t* BUF_63fdb23e9790, HASHTABLE_FIND HT_63fdb23a14e0, HASHTABLE_PROBE HT_63fdb23e9790, DBI16Type* KEY_63fdb23a14e0customer__c_city_encoded, DBI32Type* KEY_63fdb23a14e0date__d_year, DBI16Type* KEY_63fdb23a14e0supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size, DBDecimalType* lineorder__lo_revenue, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_63fdb23e9790 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_63fdb23e9790 |= reg_date__d_datekey;
//Probe Hash table
auto SLOT_63fdb23e9790 = HT_63fdb23e9790.find(KEY_63fdb23e9790);
if (SLOT_63fdb23e9790 == HT_63fdb23e9790.end()) return;
if (!(true)) return;
uint64_t KEY_63fdb23a14e0 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_63fdb23e9790[SLOT_63fdb23e9790->second * 3 + 0]];

KEY_63fdb23a14e0 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_63fdb23e9790[SLOT_63fdb23e9790->second * 3 + 2]];
KEY_63fdb23a14e0 <<= 16;
KEY_63fdb23a14e0 |= reg_supplier__s_city_encoded;
KEY_63fdb23a14e0 <<= 32;
KEY_63fdb23a14e0 |= reg_date__d_year;
//Aggregate in hashtable
auto buf_idx_63fdb23a14e0 = HT_63fdb23a14e0.find(KEY_63fdb23a14e0)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_63fdb23e9790[SLOT_63fdb23e9790->second * 3 + 1]];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_63fdb23a14e0], reg_lineorder__lo_revenue);
KEY_63fdb23a14e0customer__c_city_encoded[buf_idx_63fdb23a14e0] = reg_customer__c_city_encoded;
KEY_63fdb23a14e0supplier__s_city_encoded[buf_idx_63fdb23a14e0] = reg_supplier__s_city_encoded;
KEY_63fdb23a14e0date__d_year[buf_idx_63fdb23a14e0] = reg_date__d_year;
}
__global__ void count_63fdb240c7e0(size_t COUNT63fdb23a14e0, uint64_t* COUNT63fdb23b45d0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT63fdb23a14e0) return;
//Materialize count
atomicAdd((int*)COUNT63fdb23b45d0, 1);
}
__global__ void main_63fdb240c7e0(size_t COUNT63fdb23a14e0, DBDecimalType* MAT63fdb23b45d0aggr0__tmp_attr0, DBI16Type* MAT63fdb23b45d0customer__c_city_encoded, DBI32Type* MAT63fdb23b45d0date__d_year, DBI16Type* MAT63fdb23b45d0supplier__s_city_encoded, uint64_t* MAT_IDX63fdb23b45d0, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT63fdb23a14e0) return;
//Materialize buffers
auto mat_idx63fdb23b45d0 = atomicAdd((int*)MAT_IDX63fdb23b45d0, 1);
auto reg_customer__c_city_encoded = customer__c_city_encoded[tid];
MAT63fdb23b45d0customer__c_city_encoded[mat_idx63fdb23b45d0] = reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[tid];
MAT63fdb23b45d0supplier__s_city_encoded[mat_idx63fdb23b45d0] = reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[tid];
MAT63fdb23b45d0date__d_year[mat_idx63fdb23b45d0] = reg_date__d_year;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT63fdb23b45d0aggr0__tmp_attr0[mat_idx63fdb23b45d0] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT63fdb23f0c60;
hipMalloc(&d_COUNT63fdb23f0c60, sizeof(uint64_t));
hipMemset(d_COUNT63fdb23f0c60, 0, sizeof(uint64_t));
count_63fdb23d1120<<<std::ceil((float)customer_size/128.), 128>>>(d_COUNT63fdb23f0c60, d_customer__c_city, customer_size);
uint64_t COUNT63fdb23f0c60;
hipMemcpy(&COUNT63fdb23f0c60, d_COUNT63fdb23f0c60, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_63fdb23f0c60;
hipMalloc(&d_BUF_IDX_63fdb23f0c60, sizeof(uint64_t));
hipMemset(d_BUF_IDX_63fdb23f0c60, 0, sizeof(uint64_t));
uint64_t* d_BUF_63fdb23f0c60;
hipMalloc(&d_BUF_63fdb23f0c60, sizeof(uint64_t) * COUNT63fdb23f0c60 * 1);
auto d_HT_63fdb23f0c60 = cuco::static_map{ (int)COUNT63fdb23f0c60*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_63fdb23d1120<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_63fdb23f0c60, d_BUF_IDX_63fdb23f0c60, d_HT_63fdb23f0c60.ref(cuco::insert), d_customer__c_city, d_customer__c_custkey, customer_size);
//Materialize count
uint64_t* d_COUNT63fdb23ee570;
hipMalloc(&d_COUNT63fdb23ee570, sizeof(uint64_t));
hipMemset(d_COUNT63fdb23ee570, 0, sizeof(uint64_t));
count_63fdb23d16c0<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT63fdb23ee570, d_supplier__s_city, supplier_size);
uint64_t COUNT63fdb23ee570;
hipMemcpy(&COUNT63fdb23ee570, d_COUNT63fdb23ee570, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_63fdb23ee570;
hipMalloc(&d_BUF_IDX_63fdb23ee570, sizeof(uint64_t));
hipMemset(d_BUF_IDX_63fdb23ee570, 0, sizeof(uint64_t));
uint64_t* d_BUF_63fdb23ee570;
hipMalloc(&d_BUF_63fdb23ee570, sizeof(uint64_t) * COUNT63fdb23ee570 * 1);
auto d_HT_63fdb23ee570 = cuco::static_map{ (int)COUNT63fdb23ee570*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_63fdb23d16c0<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_63fdb23ee570, d_BUF_IDX_63fdb23ee570, d_HT_63fdb23ee570.ref(cuco::insert), d_supplier__s_city, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT63fdb23e9790;
hipMalloc(&d_COUNT63fdb23e9790, sizeof(uint64_t));
hipMemset(d_COUNT63fdb23e9790, 0, sizeof(uint64_t));
count_63fdb23eb550<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_63fdb23ee570, d_BUF_63fdb23f0c60, d_COUNT63fdb23e9790, d_HT_63fdb23ee570.ref(cuco::find), d_HT_63fdb23f0c60.ref(cuco::find), d_lineorder__lo_custkey, d_lineorder__lo_suppkey, lineorder_size);
uint64_t COUNT63fdb23e9790;
hipMemcpy(&COUNT63fdb23e9790, d_COUNT63fdb23e9790, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_63fdb23e9790;
hipMalloc(&d_BUF_IDX_63fdb23e9790, sizeof(uint64_t));
hipMemset(d_BUF_IDX_63fdb23e9790, 0, sizeof(uint64_t));
uint64_t* d_BUF_63fdb23e9790;
hipMalloc(&d_BUF_63fdb23e9790, sizeof(uint64_t) * COUNT63fdb23e9790 * 3);
auto d_HT_63fdb23e9790 = cuco::static_map{ (int)COUNT63fdb23e9790*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_63fdb23eb550<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_63fdb23e9790, d_BUF_63fdb23ee570, d_BUF_63fdb23f0c60, d_BUF_IDX_63fdb23e9790, d_HT_63fdb23e9790.ref(cuco::insert), d_HT_63fdb23ee570.ref(cuco::find), d_HT_63fdb23f0c60.ref(cuco::find), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size);
//Create aggregation hash table
auto d_HT_63fdb23a14e0 = cuco::static_map{ (int)132*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_63fdb23fd4f0<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_63fdb23e9790, d_HT_63fdb23a14e0.ref(cuco::insert), d_HT_63fdb23e9790.ref(cuco::find), d_customer__c_city_encoded, d_date__d_datekey, d_date__d_year, date_size, d_supplier__s_city_encoded);
size_t COUNT63fdb23a14e0 = d_HT_63fdb23a14e0.size();
thrust::device_vector<int64_t> keys_63fdb23a14e0(COUNT63fdb23a14e0), vals_63fdb23a14e0(COUNT63fdb23a14e0);
d_HT_63fdb23a14e0.retrieve_all(keys_63fdb23a14e0.begin(), vals_63fdb23a14e0.begin());
d_HT_63fdb23a14e0.clear();
int64_t* raw_keys63fdb23a14e0 = thrust::raw_pointer_cast(keys_63fdb23a14e0.data());
insertKeys<<<std::ceil((float)COUNT63fdb23a14e0/128.), 128>>>(raw_keys63fdb23a14e0, d_HT_63fdb23a14e0.ref(cuco::insert), COUNT63fdb23a14e0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT63fdb23a14e0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT63fdb23a14e0);
DBI16Type* d_KEY_63fdb23a14e0customer__c_city_encoded;
hipMalloc(&d_KEY_63fdb23a14e0customer__c_city_encoded, sizeof(DBI16Type) * COUNT63fdb23a14e0);
hipMemset(d_KEY_63fdb23a14e0customer__c_city_encoded, 0, sizeof(DBI16Type) * COUNT63fdb23a14e0);
DBI16Type* d_KEY_63fdb23a14e0supplier__s_city_encoded;
hipMalloc(&d_KEY_63fdb23a14e0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT63fdb23a14e0);
hipMemset(d_KEY_63fdb23a14e0supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT63fdb23a14e0);
DBI32Type* d_KEY_63fdb23a14e0date__d_year;
hipMalloc(&d_KEY_63fdb23a14e0date__d_year, sizeof(DBI32Type) * COUNT63fdb23a14e0);
hipMemset(d_KEY_63fdb23a14e0date__d_year, 0, sizeof(DBI32Type) * COUNT63fdb23a14e0);
main_63fdb23fd4f0<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_63fdb23e9790, d_HT_63fdb23a14e0.ref(cuco::find), d_HT_63fdb23e9790.ref(cuco::find), d_KEY_63fdb23a14e0customer__c_city_encoded, d_KEY_63fdb23a14e0date__d_year, d_KEY_63fdb23a14e0supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_city_encoded, d_date__d_datekey, d_date__d_year, date_size, d_lineorder__lo_revenue, d_supplier__s_city_encoded);
//Materialize count
uint64_t* d_COUNT63fdb23b45d0;
hipMalloc(&d_COUNT63fdb23b45d0, sizeof(uint64_t));
hipMemset(d_COUNT63fdb23b45d0, 0, sizeof(uint64_t));
count_63fdb240c7e0<<<std::ceil((float)COUNT63fdb23a14e0/128.), 128>>>(COUNT63fdb23a14e0, d_COUNT63fdb23b45d0);
uint64_t COUNT63fdb23b45d0;
hipMemcpy(&COUNT63fdb23b45d0, d_COUNT63fdb23b45d0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX63fdb23b45d0;
hipMalloc(&d_MAT_IDX63fdb23b45d0, sizeof(uint64_t));
hipMemset(d_MAT_IDX63fdb23b45d0, 0, sizeof(uint64_t));
auto MAT63fdb23b45d0customer__c_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT63fdb23b45d0);
DBI16Type* d_MAT63fdb23b45d0customer__c_city_encoded;
hipMalloc(&d_MAT63fdb23b45d0customer__c_city_encoded, sizeof(DBI16Type) * COUNT63fdb23b45d0);
auto MAT63fdb23b45d0supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT63fdb23b45d0);
DBI16Type* d_MAT63fdb23b45d0supplier__s_city_encoded;
hipMalloc(&d_MAT63fdb23b45d0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT63fdb23b45d0);
auto MAT63fdb23b45d0date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT63fdb23b45d0);
DBI32Type* d_MAT63fdb23b45d0date__d_year;
hipMalloc(&d_MAT63fdb23b45d0date__d_year, sizeof(DBI32Type) * COUNT63fdb23b45d0);
auto MAT63fdb23b45d0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT63fdb23b45d0);
DBDecimalType* d_MAT63fdb23b45d0aggr0__tmp_attr0;
hipMalloc(&d_MAT63fdb23b45d0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT63fdb23b45d0);
main_63fdb240c7e0<<<std::ceil((float)COUNT63fdb23a14e0/128.), 128>>>(COUNT63fdb23a14e0, d_MAT63fdb23b45d0aggr0__tmp_attr0, d_MAT63fdb23b45d0customer__c_city_encoded, d_MAT63fdb23b45d0date__d_year, d_MAT63fdb23b45d0supplier__s_city_encoded, d_MAT_IDX63fdb23b45d0, d_aggr0__tmp_attr0, d_KEY_63fdb23a14e0customer__c_city_encoded, d_KEY_63fdb23a14e0date__d_year, d_KEY_63fdb23a14e0supplier__s_city_encoded);
hipMemcpy(MAT63fdb23b45d0customer__c_city_encoded, d_MAT63fdb23b45d0customer__c_city_encoded, sizeof(DBI16Type) * COUNT63fdb23b45d0, hipMemcpyDeviceToHost);
hipMemcpy(MAT63fdb23b45d0supplier__s_city_encoded, d_MAT63fdb23b45d0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT63fdb23b45d0, hipMemcpyDeviceToHost);
hipMemcpy(MAT63fdb23b45d0date__d_year, d_MAT63fdb23b45d0date__d_year, sizeof(DBI32Type) * COUNT63fdb23b45d0, hipMemcpyDeviceToHost);
hipMemcpy(MAT63fdb23b45d0aggr0__tmp_attr0, d_MAT63fdb23b45d0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT63fdb23b45d0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT63fdb23b45d0; i++) { std::cout << "" << customer__c_city_map[MAT63fdb23b45d0customer__c_city_encoded[i]];
std::cout << "," << supplier__s_city_map[MAT63fdb23b45d0supplier__s_city_encoded[i]];
std::cout << "," << MAT63fdb23b45d0date__d_year[i];
std::cout << "," << MAT63fdb23b45d0aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_63fdb23f0c60);
hipFree(d_BUF_IDX_63fdb23f0c60);
hipFree(d_COUNT63fdb23f0c60);
hipFree(d_BUF_63fdb23ee570);
hipFree(d_BUF_IDX_63fdb23ee570);
hipFree(d_COUNT63fdb23ee570);
hipFree(d_BUF_63fdb23e9790);
hipFree(d_BUF_IDX_63fdb23e9790);
hipFree(d_COUNT63fdb23e9790);
hipFree(d_KEY_63fdb23a14e0customer__c_city_encoded);
hipFree(d_KEY_63fdb23a14e0date__d_year);
hipFree(d_KEY_63fdb23a14e0supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT63fdb23b45d0);
hipFree(d_MAT63fdb23b45d0aggr0__tmp_attr0);
hipFree(d_MAT63fdb23b45d0customer__c_city_encoded);
hipFree(d_MAT63fdb23b45d0date__d_year);
hipFree(d_MAT63fdb23b45d0supplier__s_city_encoded);
hipFree(d_MAT_IDX63fdb23b45d0);
free(MAT63fdb23b45d0aggr0__tmp_attr0);
free(MAT63fdb23b45d0customer__c_city_encoded);
free(MAT63fdb23b45d0date__d_year);
free(MAT63fdb23b45d0supplier__s_city_encoded);
}