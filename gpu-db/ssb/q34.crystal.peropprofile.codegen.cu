#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_1(uint64_t* COUNT15, DBStringType* customer__c_city, size_t customer_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_customer__c_city[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_city[ITEM] = customer__c_city[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_customer__c_city[ITEM], "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_customer__c_city[ITEM], "UNITED KI5", Predicate::eq));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT15, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_15, uint64_t* BUF_IDX_15, HASHTABLE_INSERT HT_15, DBStringType* customer__c_city, DBI32Type* customer__c_custkey, size_t customer_size, int64_t* cycles_per_warp_main_1_join_build_15, int64_t* cycles_per_warp_main_1_selection_0, int64_t* cycles_per_warp_main_1_selection_2, int64_t* cycles_per_warp_main_1_selection_3) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_customer__c_city[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_city[ITEM] = customer__c_city[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_customer__c_city[ITEM], "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_customer__c_city[ITEM], "UNITED KI5", Predicate::eq));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_3[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_15[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_15[ITEM] = 0;
KEY_15[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_15 = atomicAdd((int*)BUF_IDX_15, 1);
HT_15.insert(cuco::pair{KEY_15[ITEM], buf_idx_15});
BUF_15[(buf_idx_15) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_build_15[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_12(uint64_t* COUNT22, DBStringType* supplier__s_city, size_t supplier_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_supplier__s_city[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_city[ITEM] = supplier__s_city[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_supplier__s_city[ITEM], "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_supplier__s_city[ITEM], "UNITED KI5", Predicate::eq));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT22, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_12(uint64_t* BUF_22, uint64_t* BUF_IDX_22, HASHTABLE_INSERT HT_22, int64_t* cycles_per_warp_main_12_join_build_22, int64_t* cycles_per_warp_main_12_selection_11, int64_t* cycles_per_warp_main_12_selection_13, int64_t* cycles_per_warp_main_12_selection_14, DBStringType* supplier__s_city, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_supplier__s_city[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_city[ITEM] = supplier__s_city[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_supplier__s_city[ITEM], "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_supplier__s_city[ITEM], "UNITED KI5", Predicate::eq));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_12_selection_11[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_12_selection_13[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_12_selection_14[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_22[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_22[ITEM] = 0;
KEY_22[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_22 = atomicAdd((int*)BUF_IDX_22, 1);
HT_22.insert(cuco::pair{KEY_22[ITEM], buf_idx_22});
BUF_22[(buf_idx_22) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_12_join_build_22[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_17(uint64_t* COUNT23, DBStringType* date__d_yearmonth, size_t date_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_date__d_yearmonth[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_yearmonth[ITEM] = date__d_yearmonth[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_date__d_yearmonth[ITEM], "Dec1997", Predicate::eq);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT23, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_17(uint64_t* BUF_23, uint64_t* BUF_IDX_23, HASHTABLE_INSERT HT_23, int64_t* cycles_per_warp_main_17_join_build_23, int64_t* cycles_per_warp_main_17_selection_16, int64_t* cycles_per_warp_main_17_selection_18, int64_t* cycles_per_warp_main_17_selection_19, int64_t* cycles_per_warp_main_17_selection_20, int64_t* cycles_per_warp_main_17_selection_21, DBI32Type* date__d_datekey, DBStringType* date__d_yearmonth, size_t date_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_date__d_yearmonth[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_yearmonth[ITEM] = date__d_yearmonth[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_date__d_yearmonth[ITEM], "Dec1997", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_selection_16[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_selection_18[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_selection_19[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_selection_20[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_selection_21[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_23[ITEMS_PER_THREAD];
DBI32Type reg_date__d_datekey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_datekey[ITEM] = date__d_datekey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_23[ITEM] = 0;
KEY_23[ITEM] |= reg_date__d_datekey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_23 = atomicAdd((int*)BUF_IDX_23, 1);
HT_23.insert(cuco::pair{KEY_23[ITEM], buf_idx_23});
BUF_23[(buf_idx_23) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_join_build_23[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_5(uint64_t* BUF_15, uint64_t* BUF_22, uint64_t* BUF_23, HASHTABLE_PROBE HT_15, HASHTABLE_PROBE HT_22, HASHTABLE_PROBE HT_23, HASHTABLE_INSERT HT_24, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
uint64_t KEY_15[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_custkey[ITEM] = lineorder__lo_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_15[ITEM] = 0;
KEY_15[ITEM] |= reg_lineorder__lo_custkey[ITEM];
}
int64_t slot_second15[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_15 = HT_15.find(KEY_15[ITEM]);
if (SLOT_15 == HT_15.end()) {selection_flags[ITEM] = 0; continue;}
slot_second15[ITEM] = SLOT_15->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_22[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_suppkey[ITEM] = lineorder__lo_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_22[ITEM] = 0;
KEY_22[ITEM] |= reg_lineorder__lo_suppkey[ITEM];
}
int64_t slot_second22[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_22 = HT_22.find(KEY_22[ITEM]);
if (SLOT_22 == HT_22.end()) {selection_flags[ITEM] = 0; continue;}
slot_second22[ITEM] = SLOT_22->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_23[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_orderdate[ITEM] = lineorder__lo_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_23[ITEM] = 0;
KEY_23[ITEM] |= reg_lineorder__lo_orderdate[ITEM];
}
int64_t slot_second23[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_23 = HT_23.find(KEY_23[ITEM]);
if (SLOT_23 == HT_23.end()) {selection_flags[ITEM] = 0; continue;}
slot_second23[ITEM] = SLOT_23->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_24[ITEMS_PER_THREAD];
DBI16Type reg_customer__c_city_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_city_encoded[ITEM] = customer__c_city_encoded[BUF_15[slot_second15[ITEM] * 1 + 0]];
}
DBI16Type reg_supplier__s_city_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_supplier__s_city_encoded[ITEM] = supplier__s_city_encoded[BUF_22[slot_second22[ITEM] * 1 + 0]];
}
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_date__d_year[ITEM] = date__d_year[BUF_23[slot_second23[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_24[ITEM] = 0;
KEY_24[ITEM] |= reg_customer__c_city_encoded[ITEM];
KEY_24[ITEM] <<= 16;
KEY_24[ITEM] |= reg_supplier__s_city_encoded[ITEM];
KEY_24[ITEM] <<= 32;
KEY_24[ITEM] |= reg_date__d_year[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_24.insert(cuco::pair{KEY_24[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_5(uint64_t* BUF_15, uint64_t* BUF_22, uint64_t* BUF_23, HASHTABLE_PROBE HT_15, HASHTABLE_PROBE HT_22, HASHTABLE_PROBE HT_23, HASHTABLE_FIND HT_24, DBI16Type* KEY_24customer__c_city_encoded, DBI32Type* KEY_24date__d_year, DBI16Type* KEY_24supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, int64_t* cycles_per_warp_main_5_aggregation_24, int64_t* cycles_per_warp_main_5_join_probe_15, int64_t* cycles_per_warp_main_5_join_probe_22, int64_t* cycles_per_warp_main_5_join_probe_23, int64_t* cycles_per_warp_main_5_selection_10, int64_t* cycles_per_warp_main_5_selection_4, int64_t* cycles_per_warp_main_5_selection_6, int64_t* cycles_per_warp_main_5_selection_7, int64_t* cycles_per_warp_main_5_selection_8, int64_t* cycles_per_warp_main_5_selection_9, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_city_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_selection_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_selection_6[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_selection_7[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_selection_8[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_selection_9[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_selection_10[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_15[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_custkey[ITEM] = lineorder__lo_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_15[ITEM] = 0;
KEY_15[ITEM] |= reg_lineorder__lo_custkey[ITEM];
}
int64_t slot_second15[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_15 = HT_15.find(KEY_15[ITEM]);
if (SLOT_15 == HT_15.end()) {selection_flags[ITEM] = 0; continue;}
slot_second15[ITEM] = SLOT_15->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_join_probe_15[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_22[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_suppkey[ITEM] = lineorder__lo_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_22[ITEM] = 0;
KEY_22[ITEM] |= reg_lineorder__lo_suppkey[ITEM];
}
int64_t slot_second22[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_22 = HT_22.find(KEY_22[ITEM]);
if (SLOT_22 == HT_22.end()) {selection_flags[ITEM] = 0; continue;}
slot_second22[ITEM] = SLOT_22->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_join_probe_22[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_23[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_orderdate[ITEM] = lineorder__lo_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_23[ITEM] = 0;
KEY_23[ITEM] |= reg_lineorder__lo_orderdate[ITEM];
}
int64_t slot_second23[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_23 = HT_23.find(KEY_23[ITEM]);
if (SLOT_23 == HT_23.end()) {selection_flags[ITEM] = 0; continue;}
slot_second23[ITEM] = SLOT_23->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_join_probe_23[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_24[ITEMS_PER_THREAD];
DBI16Type reg_customer__c_city_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_city_encoded[ITEM] = customer__c_city_encoded[BUF_15[slot_second15[ITEM] * 1 + 0]];
}
DBI16Type reg_supplier__s_city_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_supplier__s_city_encoded[ITEM] = supplier__s_city_encoded[BUF_22[slot_second22[ITEM] * 1 + 0]];
}
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_date__d_year[ITEM] = date__d_year[BUF_23[slot_second23[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_24[ITEM] = 0;
KEY_24[ITEM] |= reg_customer__c_city_encoded[ITEM];
KEY_24[ITEM] <<= 16;
KEY_24[ITEM] |= reg_supplier__s_city_encoded[ITEM];
KEY_24[ITEM] <<= 32;
KEY_24[ITEM] |= reg_date__d_year[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineorder__lo_revenue[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_revenue[ITEM] = lineorder__lo_revenue[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_24 = HT_24.find(KEY_24[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr0[buf_idx_24], reg_lineorder__lo_revenue[ITEM]);
KEY_24customer__c_city_encoded[buf_idx_24] = reg_customer__c_city_encoded[ITEM];
KEY_24supplier__s_city_encoded[buf_idx_24] = reg_supplier__s_city_encoded[ITEM];
KEY_24date__d_year[buf_idx_24] = reg_date__d_year[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_aggregation_24[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_26(size_t COUNT24, uint64_t* COUNT25) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT24); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT25, 1);
}
}
__global__ void main_26(size_t COUNT24, DBDecimalType* MAT25aggr0__tmp_attr0, DBI16Type* MAT25customer__c_city_encoded, DBI32Type* MAT25date__d_year, DBI16Type* MAT25supplier__s_city_encoded, uint64_t* MAT_IDX25, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, int64_t* cycles_per_warp_main_26_materialize_25, DBI32Type* date__d_year, DBI16Type* supplier__s_city_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBI16Type reg_customer__c_city_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT24); ++ITEM) {
reg_customer__c_city_encoded[ITEM] = customer__c_city_encoded[ITEM*TB + tid];
}
DBI16Type reg_supplier__s_city_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT24); ++ITEM) {
reg_supplier__s_city_encoded[ITEM] = supplier__s_city_encoded[ITEM*TB + tid];
}
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT24); ++ITEM) {
reg_date__d_year[ITEM] = date__d_year[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT24); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT24); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx25 = atomicAdd((int*)MAT_IDX25, 1);
MAT25customer__c_city_encoded[mat_idx25] = reg_customer__c_city_encoded[ITEM];
MAT25supplier__s_city_encoded[mat_idx25] = reg_supplier__s_city_encoded[ITEM];
MAT25date__d_year[mat_idx25] = reg_date__d_year[ITEM];
MAT25aggr0__tmp_attr0[mat_idx25] = reg_aggr0__tmp_attr0[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_26_materialize_25[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_2;
auto main_1_selection_2_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_2, -1, sizeof(int64_t) * main_1_selection_2_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_3;
auto main_1_selection_3_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_3, -1, sizeof(int64_t) * main_1_selection_3_cpw_size);
//Materialize count
uint64_t* d_COUNT15;
hipMalloc(&d_COUNT15, sizeof(uint64_t));
hipMemset(d_COUNT15, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT15, d_customer__c_city, customer_size);
uint64_t COUNT15;
hipMemcpy(&COUNT15, d_COUNT15, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_1_join_build_15;
auto main_1_join_build_15_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_build_15, sizeof(int64_t) * main_1_join_build_15_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_build_15, -1, sizeof(int64_t) * main_1_join_build_15_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_15;
hipMalloc(&d_BUF_IDX_15, sizeof(uint64_t));
hipMemset(d_BUF_IDX_15, 0, sizeof(uint64_t));
uint64_t* d_BUF_15;
hipMalloc(&d_BUF_15, sizeof(uint64_t) * COUNT15 * 1);
auto d_HT_15 = cuco::static_map{ (int)COUNT15*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_15, d_BUF_IDX_15, d_HT_15.ref(cuco::insert), d_customer__c_city, d_customer__c_custkey, customer_size, d_cycles_per_warp_main_1_join_build_15, d_cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_3);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_2 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_2 ";
for (auto i=0ull; i < main_1_selection_2_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_3 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_3, d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_3 ";
for (auto i=0ull; i < main_1_selection_3_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_3[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_build_15 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_build_15_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_build_15, d_cycles_per_warp_main_1_join_build_15, sizeof(int64_t) * main_1_join_build_15_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_build_15 ";
for (auto i=0ull; i < main_1_join_build_15_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_build_15[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_12_selection_11;
auto main_12_selection_11_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_12_selection_11, sizeof(int64_t) * main_12_selection_11_cpw_size);
hipMemset(d_cycles_per_warp_main_12_selection_11, -1, sizeof(int64_t) * main_12_selection_11_cpw_size);
int64_t* d_cycles_per_warp_main_12_selection_13;
auto main_12_selection_13_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_12_selection_13, sizeof(int64_t) * main_12_selection_13_cpw_size);
hipMemset(d_cycles_per_warp_main_12_selection_13, -1, sizeof(int64_t) * main_12_selection_13_cpw_size);
int64_t* d_cycles_per_warp_main_12_selection_14;
auto main_12_selection_14_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_12_selection_14, sizeof(int64_t) * main_12_selection_14_cpw_size);
hipMemset(d_cycles_per_warp_main_12_selection_14, -1, sizeof(int64_t) * main_12_selection_14_cpw_size);
//Materialize count
uint64_t* d_COUNT22;
hipMalloc(&d_COUNT22, sizeof(uint64_t));
hipMemset(d_COUNT22, 0, sizeof(uint64_t));
count_12<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT22, d_supplier__s_city, supplier_size);
uint64_t COUNT22;
hipMemcpy(&COUNT22, d_COUNT22, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_12_join_build_22;
auto main_12_join_build_22_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_12_join_build_22, sizeof(int64_t) * main_12_join_build_22_cpw_size);
hipMemset(d_cycles_per_warp_main_12_join_build_22, -1, sizeof(int64_t) * main_12_join_build_22_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_22;
hipMalloc(&d_BUF_IDX_22, sizeof(uint64_t));
hipMemset(d_BUF_IDX_22, 0, sizeof(uint64_t));
uint64_t* d_BUF_22;
hipMalloc(&d_BUF_22, sizeof(uint64_t) * COUNT22 * 1);
auto d_HT_22 = cuco::static_map{ (int)COUNT22*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_12<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_22, d_BUF_IDX_22, d_HT_22.ref(cuco::insert), d_cycles_per_warp_main_12_join_build_22, d_cycles_per_warp_main_12_selection_11, d_cycles_per_warp_main_12_selection_13, d_cycles_per_warp_main_12_selection_14, d_supplier__s_city, d_supplier__s_suppkey, supplier_size);
int64_t* cycles_per_warp_main_12_selection_11 = (int64_t*)malloc(sizeof(int64_t) * main_12_selection_11_cpw_size);
hipMemcpy(cycles_per_warp_main_12_selection_11, d_cycles_per_warp_main_12_selection_11, sizeof(int64_t) * main_12_selection_11_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_12_selection_11 ";
for (auto i=0ull; i < main_12_selection_11_cpw_size; i++) std::cout << cycles_per_warp_main_12_selection_11[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_12_selection_13 = (int64_t*)malloc(sizeof(int64_t) * main_12_selection_13_cpw_size);
hipMemcpy(cycles_per_warp_main_12_selection_13, d_cycles_per_warp_main_12_selection_13, sizeof(int64_t) * main_12_selection_13_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_12_selection_13 ";
for (auto i=0ull; i < main_12_selection_13_cpw_size; i++) std::cout << cycles_per_warp_main_12_selection_13[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_12_selection_14 = (int64_t*)malloc(sizeof(int64_t) * main_12_selection_14_cpw_size);
hipMemcpy(cycles_per_warp_main_12_selection_14, d_cycles_per_warp_main_12_selection_14, sizeof(int64_t) * main_12_selection_14_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_12_selection_14 ";
for (auto i=0ull; i < main_12_selection_14_cpw_size; i++) std::cout << cycles_per_warp_main_12_selection_14[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_12_join_build_22 = (int64_t*)malloc(sizeof(int64_t) * main_12_join_build_22_cpw_size);
hipMemcpy(cycles_per_warp_main_12_join_build_22, d_cycles_per_warp_main_12_join_build_22, sizeof(int64_t) * main_12_join_build_22_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_12_join_build_22 ";
for (auto i=0ull; i < main_12_join_build_22_cpw_size; i++) std::cout << cycles_per_warp_main_12_join_build_22[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_17_selection_16;
auto main_17_selection_16_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_selection_16, sizeof(int64_t) * main_17_selection_16_cpw_size);
hipMemset(d_cycles_per_warp_main_17_selection_16, -1, sizeof(int64_t) * main_17_selection_16_cpw_size);
int64_t* d_cycles_per_warp_main_17_selection_18;
auto main_17_selection_18_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_selection_18, sizeof(int64_t) * main_17_selection_18_cpw_size);
hipMemset(d_cycles_per_warp_main_17_selection_18, -1, sizeof(int64_t) * main_17_selection_18_cpw_size);
int64_t* d_cycles_per_warp_main_17_selection_19;
auto main_17_selection_19_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_selection_19, sizeof(int64_t) * main_17_selection_19_cpw_size);
hipMemset(d_cycles_per_warp_main_17_selection_19, -1, sizeof(int64_t) * main_17_selection_19_cpw_size);
int64_t* d_cycles_per_warp_main_17_selection_20;
auto main_17_selection_20_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_selection_20, sizeof(int64_t) * main_17_selection_20_cpw_size);
hipMemset(d_cycles_per_warp_main_17_selection_20, -1, sizeof(int64_t) * main_17_selection_20_cpw_size);
int64_t* d_cycles_per_warp_main_17_selection_21;
auto main_17_selection_21_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_selection_21, sizeof(int64_t) * main_17_selection_21_cpw_size);
hipMemset(d_cycles_per_warp_main_17_selection_21, -1, sizeof(int64_t) * main_17_selection_21_cpw_size);
//Materialize count
uint64_t* d_COUNT23;
hipMalloc(&d_COUNT23, sizeof(uint64_t));
hipMemset(d_COUNT23, 0, sizeof(uint64_t));
count_17<<<std::ceil((float)date_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT23, d_date__d_yearmonth, date_size);
uint64_t COUNT23;
hipMemcpy(&COUNT23, d_COUNT23, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_17_join_build_23;
auto main_17_join_build_23_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_join_build_23, sizeof(int64_t) * main_17_join_build_23_cpw_size);
hipMemset(d_cycles_per_warp_main_17_join_build_23, -1, sizeof(int64_t) * main_17_join_build_23_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_23;
hipMalloc(&d_BUF_IDX_23, sizeof(uint64_t));
hipMemset(d_BUF_IDX_23, 0, sizeof(uint64_t));
uint64_t* d_BUF_23;
hipMalloc(&d_BUF_23, sizeof(uint64_t) * COUNT23 * 1);
auto d_HT_23 = cuco::static_map{ (int)COUNT23*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_17<<<std::ceil((float)date_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_23, d_BUF_IDX_23, d_HT_23.ref(cuco::insert), d_cycles_per_warp_main_17_join_build_23, d_cycles_per_warp_main_17_selection_16, d_cycles_per_warp_main_17_selection_18, d_cycles_per_warp_main_17_selection_19, d_cycles_per_warp_main_17_selection_20, d_cycles_per_warp_main_17_selection_21, d_date__d_datekey, d_date__d_yearmonth, date_size);
int64_t* cycles_per_warp_main_17_selection_16 = (int64_t*)malloc(sizeof(int64_t) * main_17_selection_16_cpw_size);
hipMemcpy(cycles_per_warp_main_17_selection_16, d_cycles_per_warp_main_17_selection_16, sizeof(int64_t) * main_17_selection_16_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_selection_16 ";
for (auto i=0ull; i < main_17_selection_16_cpw_size; i++) std::cout << cycles_per_warp_main_17_selection_16[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_17_selection_18 = (int64_t*)malloc(sizeof(int64_t) * main_17_selection_18_cpw_size);
hipMemcpy(cycles_per_warp_main_17_selection_18, d_cycles_per_warp_main_17_selection_18, sizeof(int64_t) * main_17_selection_18_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_selection_18 ";
for (auto i=0ull; i < main_17_selection_18_cpw_size; i++) std::cout << cycles_per_warp_main_17_selection_18[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_17_selection_19 = (int64_t*)malloc(sizeof(int64_t) * main_17_selection_19_cpw_size);
hipMemcpy(cycles_per_warp_main_17_selection_19, d_cycles_per_warp_main_17_selection_19, sizeof(int64_t) * main_17_selection_19_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_selection_19 ";
for (auto i=0ull; i < main_17_selection_19_cpw_size; i++) std::cout << cycles_per_warp_main_17_selection_19[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_17_selection_20 = (int64_t*)malloc(sizeof(int64_t) * main_17_selection_20_cpw_size);
hipMemcpy(cycles_per_warp_main_17_selection_20, d_cycles_per_warp_main_17_selection_20, sizeof(int64_t) * main_17_selection_20_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_selection_20 ";
for (auto i=0ull; i < main_17_selection_20_cpw_size; i++) std::cout << cycles_per_warp_main_17_selection_20[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_17_selection_21 = (int64_t*)malloc(sizeof(int64_t) * main_17_selection_21_cpw_size);
hipMemcpy(cycles_per_warp_main_17_selection_21, d_cycles_per_warp_main_17_selection_21, sizeof(int64_t) * main_17_selection_21_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_selection_21 ";
for (auto i=0ull; i < main_17_selection_21_cpw_size; i++) std::cout << cycles_per_warp_main_17_selection_21[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_17_join_build_23 = (int64_t*)malloc(sizeof(int64_t) * main_17_join_build_23_cpw_size);
hipMemcpy(cycles_per_warp_main_17_join_build_23, d_cycles_per_warp_main_17_join_build_23, sizeof(int64_t) * main_17_join_build_23_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_join_build_23 ";
for (auto i=0ull; i < main_17_join_build_23_cpw_size; i++) std::cout << cycles_per_warp_main_17_join_build_23[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_5_selection_4;
auto main_5_selection_4_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_selection_4, sizeof(int64_t) * main_5_selection_4_cpw_size);
hipMemset(d_cycles_per_warp_main_5_selection_4, -1, sizeof(int64_t) * main_5_selection_4_cpw_size);
int64_t* d_cycles_per_warp_main_5_selection_6;
auto main_5_selection_6_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_selection_6, sizeof(int64_t) * main_5_selection_6_cpw_size);
hipMemset(d_cycles_per_warp_main_5_selection_6, -1, sizeof(int64_t) * main_5_selection_6_cpw_size);
int64_t* d_cycles_per_warp_main_5_selection_7;
auto main_5_selection_7_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_selection_7, sizeof(int64_t) * main_5_selection_7_cpw_size);
hipMemset(d_cycles_per_warp_main_5_selection_7, -1, sizeof(int64_t) * main_5_selection_7_cpw_size);
int64_t* d_cycles_per_warp_main_5_selection_8;
auto main_5_selection_8_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_selection_8, sizeof(int64_t) * main_5_selection_8_cpw_size);
hipMemset(d_cycles_per_warp_main_5_selection_8, -1, sizeof(int64_t) * main_5_selection_8_cpw_size);
int64_t* d_cycles_per_warp_main_5_selection_9;
auto main_5_selection_9_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_selection_9, sizeof(int64_t) * main_5_selection_9_cpw_size);
hipMemset(d_cycles_per_warp_main_5_selection_9, -1, sizeof(int64_t) * main_5_selection_9_cpw_size);
int64_t* d_cycles_per_warp_main_5_selection_10;
auto main_5_selection_10_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_selection_10, sizeof(int64_t) * main_5_selection_10_cpw_size);
hipMemset(d_cycles_per_warp_main_5_selection_10, -1, sizeof(int64_t) * main_5_selection_10_cpw_size);
int64_t* d_cycles_per_warp_main_5_join_probe_15;
auto main_5_join_probe_15_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_join_probe_15, sizeof(int64_t) * main_5_join_probe_15_cpw_size);
hipMemset(d_cycles_per_warp_main_5_join_probe_15, -1, sizeof(int64_t) * main_5_join_probe_15_cpw_size);
int64_t* d_cycles_per_warp_main_5_join_probe_22;
auto main_5_join_probe_22_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_join_probe_22, sizeof(int64_t) * main_5_join_probe_22_cpw_size);
hipMemset(d_cycles_per_warp_main_5_join_probe_22, -1, sizeof(int64_t) * main_5_join_probe_22_cpw_size);
int64_t* d_cycles_per_warp_main_5_join_probe_23;
auto main_5_join_probe_23_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_join_probe_23, sizeof(int64_t) * main_5_join_probe_23_cpw_size);
hipMemset(d_cycles_per_warp_main_5_join_probe_23, -1, sizeof(int64_t) * main_5_join_probe_23_cpw_size);
//Create aggregation hash table
auto d_HT_24 = cuco::static_map{ (int)3*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5<<<std::ceil((float)lineorder_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_15, d_BUF_22, d_BUF_23, d_HT_15.ref(cuco::find), d_HT_22.ref(cuco::find), d_HT_23.ref(cuco::find), d_HT_24.ref(cuco::insert), d_customer__c_city_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
size_t COUNT24 = d_HT_24.size();
thrust::device_vector<int64_t> keys_24(COUNT24), vals_24(COUNT24);
d_HT_24.retrieve_all(keys_24.begin(), vals_24.begin());
d_HT_24.clear();
int64_t* raw_keys24 = thrust::raw_pointer_cast(keys_24.data());
insertKeys<<<std::ceil((float)COUNT24/128.), 128>>>(raw_keys24, d_HT_24.ref(cuco::insert), COUNT24);
int64_t* d_cycles_per_warp_main_5_aggregation_24;
auto main_5_aggregation_24_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_aggregation_24, sizeof(int64_t) * main_5_aggregation_24_cpw_size);
hipMemset(d_cycles_per_warp_main_5_aggregation_24, -1, sizeof(int64_t) * main_5_aggregation_24_cpw_size);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT24);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT24);
DBI16Type* d_KEY_24customer__c_city_encoded;
hipMalloc(&d_KEY_24customer__c_city_encoded, sizeof(DBI16Type) * COUNT24);
hipMemset(d_KEY_24customer__c_city_encoded, 0, sizeof(DBI16Type) * COUNT24);
DBI16Type* d_KEY_24supplier__s_city_encoded;
hipMalloc(&d_KEY_24supplier__s_city_encoded, sizeof(DBI16Type) * COUNT24);
hipMemset(d_KEY_24supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT24);
DBI32Type* d_KEY_24date__d_year;
hipMalloc(&d_KEY_24date__d_year, sizeof(DBI32Type) * COUNT24);
hipMemset(d_KEY_24date__d_year, 0, sizeof(DBI32Type) * COUNT24);
main_5<<<std::ceil((float)lineorder_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_15, d_BUF_22, d_BUF_23, d_HT_15.ref(cuco::find), d_HT_22.ref(cuco::find), d_HT_23.ref(cuco::find), d_HT_24.ref(cuco::find), d_KEY_24customer__c_city_encoded, d_KEY_24date__d_year, d_KEY_24supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_city_encoded, d_cycles_per_warp_main_5_aggregation_24, d_cycles_per_warp_main_5_join_probe_15, d_cycles_per_warp_main_5_join_probe_22, d_cycles_per_warp_main_5_join_probe_23, d_cycles_per_warp_main_5_selection_10, d_cycles_per_warp_main_5_selection_4, d_cycles_per_warp_main_5_selection_6, d_cycles_per_warp_main_5_selection_7, d_cycles_per_warp_main_5_selection_8, d_cycles_per_warp_main_5_selection_9, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_city_encoded);
int64_t* cycles_per_warp_main_5_selection_4 = (int64_t*)malloc(sizeof(int64_t) * main_5_selection_4_cpw_size);
hipMemcpy(cycles_per_warp_main_5_selection_4, d_cycles_per_warp_main_5_selection_4, sizeof(int64_t) * main_5_selection_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_selection_4 ";
for (auto i=0ull; i < main_5_selection_4_cpw_size; i++) std::cout << cycles_per_warp_main_5_selection_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_selection_6 = (int64_t*)malloc(sizeof(int64_t) * main_5_selection_6_cpw_size);
hipMemcpy(cycles_per_warp_main_5_selection_6, d_cycles_per_warp_main_5_selection_6, sizeof(int64_t) * main_5_selection_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_selection_6 ";
for (auto i=0ull; i < main_5_selection_6_cpw_size; i++) std::cout << cycles_per_warp_main_5_selection_6[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_selection_7 = (int64_t*)malloc(sizeof(int64_t) * main_5_selection_7_cpw_size);
hipMemcpy(cycles_per_warp_main_5_selection_7, d_cycles_per_warp_main_5_selection_7, sizeof(int64_t) * main_5_selection_7_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_selection_7 ";
for (auto i=0ull; i < main_5_selection_7_cpw_size; i++) std::cout << cycles_per_warp_main_5_selection_7[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_selection_8 = (int64_t*)malloc(sizeof(int64_t) * main_5_selection_8_cpw_size);
hipMemcpy(cycles_per_warp_main_5_selection_8, d_cycles_per_warp_main_5_selection_8, sizeof(int64_t) * main_5_selection_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_selection_8 ";
for (auto i=0ull; i < main_5_selection_8_cpw_size; i++) std::cout << cycles_per_warp_main_5_selection_8[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_selection_9 = (int64_t*)malloc(sizeof(int64_t) * main_5_selection_9_cpw_size);
hipMemcpy(cycles_per_warp_main_5_selection_9, d_cycles_per_warp_main_5_selection_9, sizeof(int64_t) * main_5_selection_9_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_selection_9 ";
for (auto i=0ull; i < main_5_selection_9_cpw_size; i++) std::cout << cycles_per_warp_main_5_selection_9[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_selection_10 = (int64_t*)malloc(sizeof(int64_t) * main_5_selection_10_cpw_size);
hipMemcpy(cycles_per_warp_main_5_selection_10, d_cycles_per_warp_main_5_selection_10, sizeof(int64_t) * main_5_selection_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_selection_10 ";
for (auto i=0ull; i < main_5_selection_10_cpw_size; i++) std::cout << cycles_per_warp_main_5_selection_10[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_join_probe_15 = (int64_t*)malloc(sizeof(int64_t) * main_5_join_probe_15_cpw_size);
hipMemcpy(cycles_per_warp_main_5_join_probe_15, d_cycles_per_warp_main_5_join_probe_15, sizeof(int64_t) * main_5_join_probe_15_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_join_probe_15 ";
for (auto i=0ull; i < main_5_join_probe_15_cpw_size; i++) std::cout << cycles_per_warp_main_5_join_probe_15[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_join_probe_22 = (int64_t*)malloc(sizeof(int64_t) * main_5_join_probe_22_cpw_size);
hipMemcpy(cycles_per_warp_main_5_join_probe_22, d_cycles_per_warp_main_5_join_probe_22, sizeof(int64_t) * main_5_join_probe_22_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_join_probe_22 ";
for (auto i=0ull; i < main_5_join_probe_22_cpw_size; i++) std::cout << cycles_per_warp_main_5_join_probe_22[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_join_probe_23 = (int64_t*)malloc(sizeof(int64_t) * main_5_join_probe_23_cpw_size);
hipMemcpy(cycles_per_warp_main_5_join_probe_23, d_cycles_per_warp_main_5_join_probe_23, sizeof(int64_t) * main_5_join_probe_23_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_join_probe_23 ";
for (auto i=0ull; i < main_5_join_probe_23_cpw_size; i++) std::cout << cycles_per_warp_main_5_join_probe_23[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_aggregation_24 = (int64_t*)malloc(sizeof(int64_t) * main_5_aggregation_24_cpw_size);
hipMemcpy(cycles_per_warp_main_5_aggregation_24, d_cycles_per_warp_main_5_aggregation_24, sizeof(int64_t) * main_5_aggregation_24_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_aggregation_24 ";
for (auto i=0ull; i < main_5_aggregation_24_cpw_size; i++) std::cout << cycles_per_warp_main_5_aggregation_24[i] << " ";
std::cout << std::endl;
//Materialize count
uint64_t* d_COUNT25;
hipMalloc(&d_COUNT25, sizeof(uint64_t));
hipMemset(d_COUNT25, 0, sizeof(uint64_t));
count_26<<<std::ceil((float)COUNT24/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT24, d_COUNT25);
uint64_t COUNT25;
hipMemcpy(&COUNT25, d_COUNT25, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_26_materialize_25;
auto main_26_materialize_25_cpw_size = std::ceil((float)COUNT24/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_26_materialize_25, sizeof(int64_t) * main_26_materialize_25_cpw_size);
hipMemset(d_cycles_per_warp_main_26_materialize_25, -1, sizeof(int64_t) * main_26_materialize_25_cpw_size);
//Materialize buffers
uint64_t* d_MAT_IDX25;
hipMalloc(&d_MAT_IDX25, sizeof(uint64_t));
hipMemset(d_MAT_IDX25, 0, sizeof(uint64_t));
auto MAT25customer__c_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT25);
DBI16Type* d_MAT25customer__c_city_encoded;
hipMalloc(&d_MAT25customer__c_city_encoded, sizeof(DBI16Type) * COUNT25);
auto MAT25supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT25);
DBI16Type* d_MAT25supplier__s_city_encoded;
hipMalloc(&d_MAT25supplier__s_city_encoded, sizeof(DBI16Type) * COUNT25);
auto MAT25date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT25);
DBI32Type* d_MAT25date__d_year;
hipMalloc(&d_MAT25date__d_year, sizeof(DBI32Type) * COUNT25);
auto MAT25aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT25);
DBDecimalType* d_MAT25aggr0__tmp_attr0;
hipMalloc(&d_MAT25aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT25);
main_26<<<std::ceil((float)COUNT24/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT24, d_MAT25aggr0__tmp_attr0, d_MAT25customer__c_city_encoded, d_MAT25date__d_year, d_MAT25supplier__s_city_encoded, d_MAT_IDX25, d_aggr0__tmp_attr0, d_KEY_24customer__c_city_encoded, d_cycles_per_warp_main_26_materialize_25, d_KEY_24date__d_year, d_KEY_24supplier__s_city_encoded);
hipMemcpy(MAT25customer__c_city_encoded, d_MAT25customer__c_city_encoded, sizeof(DBI16Type) * COUNT25, hipMemcpyDeviceToHost);
hipMemcpy(MAT25supplier__s_city_encoded, d_MAT25supplier__s_city_encoded, sizeof(DBI16Type) * COUNT25, hipMemcpyDeviceToHost);
hipMemcpy(MAT25date__d_year, d_MAT25date__d_year, sizeof(DBI32Type) * COUNT25, hipMemcpyDeviceToHost);
hipMemcpy(MAT25aggr0__tmp_attr0, d_MAT25aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT25, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_26_materialize_25 = (int64_t*)malloc(sizeof(int64_t) * main_26_materialize_25_cpw_size);
hipMemcpy(cycles_per_warp_main_26_materialize_25, d_cycles_per_warp_main_26_materialize_25, sizeof(int64_t) * main_26_materialize_25_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_26_materialize_25 ";
for (auto i=0ull; i < main_26_materialize_25_cpw_size; i++) std::cout << cycles_per_warp_main_26_materialize_25[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
      size_t aux_mem = usedGpuMem() - used_mem;
      std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_15);
hipFree(d_BUF_IDX_15);
hipFree(d_COUNT15);
hipFree(d_BUF_22);
hipFree(d_BUF_IDX_22);
hipFree(d_COUNT22);
hipFree(d_BUF_23);
hipFree(d_BUF_IDX_23);
hipFree(d_COUNT23);
hipFree(d_KEY_24customer__c_city_encoded);
hipFree(d_KEY_24date__d_year);
hipFree(d_KEY_24supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT25);
hipFree(d_MAT25aggr0__tmp_attr0);
hipFree(d_MAT25customer__c_city_encoded);
hipFree(d_MAT25date__d_year);
hipFree(d_MAT25supplier__s_city_encoded);
hipFree(d_MAT_IDX25);
free(MAT25aggr0__tmp_attr0);
free(MAT25customer__c_city_encoded);
free(MAT25date__d_year);
free(MAT25supplier__s_city_encoded);
}