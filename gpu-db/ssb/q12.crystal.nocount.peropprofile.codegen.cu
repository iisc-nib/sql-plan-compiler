#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_INSERT>
__global__ void main_10(uint64_t* BUF_15, uint64_t* BUF_IDX_15, HASHTABLE_INSERT HT_15, int64_t* cycles_per_warp_main_10_join_build_15, int64_t* cycles_per_warp_main_10_selection_11, int64_t* cycles_per_warp_main_10_selection_12, int64_t* cycles_per_warp_main_10_selection_13, int64_t* cycles_per_warp_main_10_selection_14, int64_t* cycles_per_warp_main_10_selection_9, DBI32Type* date__d_datekey, DBI32Type* date__d_yearmonthnum, size_t date_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBI32Type reg_date__d_yearmonthnum[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_yearmonthnum[ITEM] = date__d_yearmonthnum[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_date__d_yearmonthnum[ITEM], 199401, Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_10_selection_9[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_10_selection_11[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_10_selection_12[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_10_selection_13[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_10_selection_14[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_15[ITEMS_PER_THREAD];
DBI32Type reg_date__d_datekey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_datekey[ITEM] = date__d_datekey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_15[ITEM] = 0;
KEY_15[ITEM] |= reg_date__d_datekey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_15.insert(cuco::pair{KEY_15[ITEM], ITEM*TB + tid});
BUF_15[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_10_join_build_15[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_1(uint64_t* BUF_15, HASHTABLE_PROBE HT_15, HASHTABLE_FIND HT_17, int* SLOT_COUNT_17, DBDecimalType* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_1_aggregation_17, int64_t* cycles_per_warp_main_1_join_probe_15, int64_t* cycles_per_warp_main_1_map_16, int64_t* cycles_per_warp_main_1_selection_0, int64_t* cycles_per_warp_main_1_selection_2, int64_t* cycles_per_warp_main_1_selection_3, int64_t* cycles_per_warp_main_1_selection_4, int64_t* cycles_per_warp_main_1_selection_5, int64_t* cycles_per_warp_main_1_selection_6, int64_t* cycles_per_warp_main_1_selection_7, int64_t* cycles_per_warp_main_1_selection_8, DBI32Type* lineorder__lo_discount, DBDecimalType* lineorder__lo_extendedprice, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBI32Type reg_lineorder__lo_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_discount[ITEM] = lineorder__lo_discount[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineorder__lo_discount[ITEM], 4, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount[ITEM], 6, Predicate::lte);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
DBI32Type reg_lineorder__lo_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_quantity[ITEM] = lineorder__lo_quantity[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineorder__lo_quantity[ITEM], 26, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_quantity[ITEM], 35, Predicate::lte);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_3[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_5[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_6[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_7[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_8[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_15[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_orderdate[ITEM] = lineorder__lo_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_15[ITEM] = 0;
KEY_15[ITEM] |= reg_lineorder__lo_orderdate[ITEM];
}
//Probe Hash table
int64_t slot_second15[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_15 = HT_15.find(KEY_15[ITEM]);
if (SLOT_15 == HT_15.end()) {selection_flags[ITEM] = 0; continue;}
slot_second15[ITEM] = SLOT_15->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_probe_15[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_map_16[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_17[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_17[ITEM] = 0;
}
//Aggregate in hashtable
DBDecimalType reg_lineorder__lo_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_extendedprice[ITEM] = lineorder__lo_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineorder__lo_extendedprice[ITEM]) * ((DBDecimalType)(reg_lineorder__lo_discount[ITEM]));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_17 = get_aggregation_slot(KEY_17[ITEM], HT_17, SLOT_COUNT_17);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_17], reg_map0__tmp_attr1[ITEM]);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_aggregation_17[blockIdx.x] = cycles_per_warp;}
}
__global__ void main_19(size_t COUNT17, DBDecimalType* MAT18aggr0__tmp_attr0, uint64_t* MAT_IDX18, DBDecimalType* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_19_materialize_18) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT17); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT17); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx18 = atomicAdd((int*)MAT_IDX18, 1);
MAT18aggr0__tmp_attr0[mat_idx18] = reg_aggr0__tmp_attr0[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_19_materialize_18[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_10_selection_9;
auto main_10_selection_9_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_10_selection_9, sizeof(int64_t) * main_10_selection_9_cpw_size);
hipMemset(d_cycles_per_warp_main_10_selection_9, -1, sizeof(int64_t) * main_10_selection_9_cpw_size);
int64_t* d_cycles_per_warp_main_10_selection_11;
auto main_10_selection_11_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_10_selection_11, sizeof(int64_t) * main_10_selection_11_cpw_size);
hipMemset(d_cycles_per_warp_main_10_selection_11, -1, sizeof(int64_t) * main_10_selection_11_cpw_size);
int64_t* d_cycles_per_warp_main_10_selection_12;
auto main_10_selection_12_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_10_selection_12, sizeof(int64_t) * main_10_selection_12_cpw_size);
hipMemset(d_cycles_per_warp_main_10_selection_12, -1, sizeof(int64_t) * main_10_selection_12_cpw_size);
int64_t* d_cycles_per_warp_main_10_selection_13;
auto main_10_selection_13_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_10_selection_13, sizeof(int64_t) * main_10_selection_13_cpw_size);
hipMemset(d_cycles_per_warp_main_10_selection_13, -1, sizeof(int64_t) * main_10_selection_13_cpw_size);
int64_t* d_cycles_per_warp_main_10_selection_14;
auto main_10_selection_14_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_10_selection_14, sizeof(int64_t) * main_10_selection_14_cpw_size);
hipMemset(d_cycles_per_warp_main_10_selection_14, -1, sizeof(int64_t) * main_10_selection_14_cpw_size);
int64_t* d_cycles_per_warp_main_10_join_build_15;
auto main_10_join_build_15_cpw_size = std::ceil((float)date_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_10_join_build_15, sizeof(int64_t) * main_10_join_build_15_cpw_size);
hipMemset(d_cycles_per_warp_main_10_join_build_15, -1, sizeof(int64_t) * main_10_join_build_15_cpw_size);
size_t COUNT15 = date_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_15;
hipMalloc(&d_BUF_IDX_15, sizeof(uint64_t));
hipMemset(d_BUF_IDX_15, 0, sizeof(uint64_t));
uint64_t* d_BUF_15;
hipMalloc(&d_BUF_15, sizeof(uint64_t) * COUNT15 * 1);
auto d_HT_15 = cuco::static_map{ (int)COUNT15*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_10<<<std::ceil((float)date_size/(float)TILE_SIZE), TB>>>(d_BUF_15, d_BUF_IDX_15, d_HT_15.ref(cuco::insert), d_cycles_per_warp_main_10_join_build_15, d_cycles_per_warp_main_10_selection_11, d_cycles_per_warp_main_10_selection_12, d_cycles_per_warp_main_10_selection_13, d_cycles_per_warp_main_10_selection_14, d_cycles_per_warp_main_10_selection_9, d_date__d_datekey, d_date__d_yearmonthnum, date_size);
int64_t* cycles_per_warp_main_10_selection_9 = (int64_t*)malloc(sizeof(int64_t) * main_10_selection_9_cpw_size);
hipMemcpy(cycles_per_warp_main_10_selection_9, d_cycles_per_warp_main_10_selection_9, sizeof(int64_t) * main_10_selection_9_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_10_selection_9 ";
for (auto i=0ull; i < main_10_selection_9_cpw_size; i++) std::cout << cycles_per_warp_main_10_selection_9[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_10_selection_11 = (int64_t*)malloc(sizeof(int64_t) * main_10_selection_11_cpw_size);
hipMemcpy(cycles_per_warp_main_10_selection_11, d_cycles_per_warp_main_10_selection_11, sizeof(int64_t) * main_10_selection_11_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_10_selection_11 ";
for (auto i=0ull; i < main_10_selection_11_cpw_size; i++) std::cout << cycles_per_warp_main_10_selection_11[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_10_selection_12 = (int64_t*)malloc(sizeof(int64_t) * main_10_selection_12_cpw_size);
hipMemcpy(cycles_per_warp_main_10_selection_12, d_cycles_per_warp_main_10_selection_12, sizeof(int64_t) * main_10_selection_12_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_10_selection_12 ";
for (auto i=0ull; i < main_10_selection_12_cpw_size; i++) std::cout << cycles_per_warp_main_10_selection_12[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_10_selection_13 = (int64_t*)malloc(sizeof(int64_t) * main_10_selection_13_cpw_size);
hipMemcpy(cycles_per_warp_main_10_selection_13, d_cycles_per_warp_main_10_selection_13, sizeof(int64_t) * main_10_selection_13_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_10_selection_13 ";
for (auto i=0ull; i < main_10_selection_13_cpw_size; i++) std::cout << cycles_per_warp_main_10_selection_13[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_10_selection_14 = (int64_t*)malloc(sizeof(int64_t) * main_10_selection_14_cpw_size);
hipMemcpy(cycles_per_warp_main_10_selection_14, d_cycles_per_warp_main_10_selection_14, sizeof(int64_t) * main_10_selection_14_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_10_selection_14 ";
for (auto i=0ull; i < main_10_selection_14_cpw_size; i++) std::cout << cycles_per_warp_main_10_selection_14[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_10_join_build_15 = (int64_t*)malloc(sizeof(int64_t) * main_10_join_build_15_cpw_size);
hipMemcpy(cycles_per_warp_main_10_join_build_15, d_cycles_per_warp_main_10_join_build_15, sizeof(int64_t) * main_10_join_build_15_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_10_join_build_15 ";
for (auto i=0ull; i < main_10_join_build_15_cpw_size; i++) std::cout << cycles_per_warp_main_10_join_build_15[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_2;
auto main_1_selection_2_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_2, -1, sizeof(int64_t) * main_1_selection_2_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_3;
auto main_1_selection_3_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_3, -1, sizeof(int64_t) * main_1_selection_3_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_4;
auto main_1_selection_4_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_4, sizeof(int64_t) * main_1_selection_4_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_4, -1, sizeof(int64_t) * main_1_selection_4_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_5;
auto main_1_selection_5_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_5, sizeof(int64_t) * main_1_selection_5_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_5, -1, sizeof(int64_t) * main_1_selection_5_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_6;
auto main_1_selection_6_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_6, sizeof(int64_t) * main_1_selection_6_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_6, -1, sizeof(int64_t) * main_1_selection_6_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_7;
auto main_1_selection_7_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_7, sizeof(int64_t) * main_1_selection_7_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_7, -1, sizeof(int64_t) * main_1_selection_7_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_8;
auto main_1_selection_8_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_8, sizeof(int64_t) * main_1_selection_8_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_8, -1, sizeof(int64_t) * main_1_selection_8_cpw_size);
int64_t* d_cycles_per_warp_main_1_join_probe_15;
auto main_1_join_probe_15_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_probe_15, sizeof(int64_t) * main_1_join_probe_15_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_probe_15, -1, sizeof(int64_t) * main_1_join_probe_15_cpw_size);
int64_t* d_cycles_per_warp_main_1_map_16;
auto main_1_map_16_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_map_16, sizeof(int64_t) * main_1_map_16_cpw_size);
hipMemset(d_cycles_per_warp_main_1_map_16, -1, sizeof(int64_t) * main_1_map_16_cpw_size);
int64_t* d_cycles_per_warp_main_1_aggregation_17;
auto main_1_aggregation_17_cpw_size = std::ceil((float)lineorder_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_aggregation_17, sizeof(int64_t) * main_1_aggregation_17_cpw_size);
hipMemset(d_cycles_per_warp_main_1_aggregation_17, -1, sizeof(int64_t) * main_1_aggregation_17_cpw_size);
size_t COUNT17 = 1;
auto d_HT_17 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_17;
hipMalloc(&d_SLOT_COUNT_17, sizeof(int));
hipMemset(d_SLOT_COUNT_17, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT17);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT17);
main_1<<<std::ceil((float)lineorder_size/(float)TILE_SIZE), TB>>>(d_BUF_15, d_HT_15.ref(cuco::find), d_HT_17.ref(cuco::insert_and_find), d_SLOT_COUNT_17, d_aggr0__tmp_attr0, d_cycles_per_warp_main_1_aggregation_17, d_cycles_per_warp_main_1_join_probe_15, d_cycles_per_warp_main_1_map_16, d_cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_3, d_cycles_per_warp_main_1_selection_4, d_cycles_per_warp_main_1_selection_5, d_cycles_per_warp_main_1_selection_6, d_cycles_per_warp_main_1_selection_7, d_cycles_per_warp_main_1_selection_8, d_lineorder__lo_discount, d_lineorder__lo_extendedprice, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_2 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_2 ";
for (auto i=0ull; i < main_1_selection_2_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_3 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_3, d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_3 ";
for (auto i=0ull; i < main_1_selection_3_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_3[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_4 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_4_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_4, d_cycles_per_warp_main_1_selection_4, sizeof(int64_t) * main_1_selection_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_4 ";
for (auto i=0ull; i < main_1_selection_4_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_5 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_5_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_5, d_cycles_per_warp_main_1_selection_5, sizeof(int64_t) * main_1_selection_5_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_5 ";
for (auto i=0ull; i < main_1_selection_5_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_5[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_6 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_6_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_6, d_cycles_per_warp_main_1_selection_6, sizeof(int64_t) * main_1_selection_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_6 ";
for (auto i=0ull; i < main_1_selection_6_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_6[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_7 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_7_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_7, d_cycles_per_warp_main_1_selection_7, sizeof(int64_t) * main_1_selection_7_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_7 ";
for (auto i=0ull; i < main_1_selection_7_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_7[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_8 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_8_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_8, d_cycles_per_warp_main_1_selection_8, sizeof(int64_t) * main_1_selection_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_8 ";
for (auto i=0ull; i < main_1_selection_8_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_8[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_probe_15 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_probe_15_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_probe_15, d_cycles_per_warp_main_1_join_probe_15, sizeof(int64_t) * main_1_join_probe_15_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_probe_15 ";
for (auto i=0ull; i < main_1_join_probe_15_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_probe_15[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_map_16 = (int64_t*)malloc(sizeof(int64_t) * main_1_map_16_cpw_size);
hipMemcpy(cycles_per_warp_main_1_map_16, d_cycles_per_warp_main_1_map_16, sizeof(int64_t) * main_1_map_16_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_map_16 ";
for (auto i=0ull; i < main_1_map_16_cpw_size; i++) std::cout << cycles_per_warp_main_1_map_16[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_aggregation_17 = (int64_t*)malloc(sizeof(int64_t) * main_1_aggregation_17_cpw_size);
hipMemcpy(cycles_per_warp_main_1_aggregation_17, d_cycles_per_warp_main_1_aggregation_17, sizeof(int64_t) * main_1_aggregation_17_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_aggregation_17 ";
for (auto i=0ull; i < main_1_aggregation_17_cpw_size; i++) std::cout << cycles_per_warp_main_1_aggregation_17[i] << " ";
std::cout << std::endl;
COUNT17 = d_HT_17.size();
int64_t* d_cycles_per_warp_main_19_materialize_18;
auto main_19_materialize_18_cpw_size = std::ceil((float)COUNT17/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_19_materialize_18, sizeof(int64_t) * main_19_materialize_18_cpw_size);
hipMemset(d_cycles_per_warp_main_19_materialize_18, -1, sizeof(int64_t) * main_19_materialize_18_cpw_size);
size_t COUNT18 = COUNT17;
//Materialize buffers
uint64_t* d_MAT_IDX18;
hipMalloc(&d_MAT_IDX18, sizeof(uint64_t));
hipMemset(d_MAT_IDX18, 0, sizeof(uint64_t));
auto MAT18aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT18);
DBDecimalType* d_MAT18aggr0__tmp_attr0;
hipMalloc(&d_MAT18aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT18);
main_19<<<std::ceil((float)COUNT17/(float)TILE_SIZE), TB>>>(COUNT17, d_MAT18aggr0__tmp_attr0, d_MAT_IDX18, d_aggr0__tmp_attr0, d_cycles_per_warp_main_19_materialize_18);
uint64_t MATCOUNT_18 = 0;
hipMemcpy(&MATCOUNT_18, d_MAT_IDX18, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT18aggr0__tmp_attr0, d_MAT18aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT18, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_19_materialize_18 = (int64_t*)malloc(sizeof(int64_t) * main_19_materialize_18_cpw_size);
hipMemcpy(cycles_per_warp_main_19_materialize_18, d_cycles_per_warp_main_19_materialize_18, sizeof(int64_t) * main_19_materialize_18_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_19_materialize_18 ";
for (auto i=0ull; i < main_19_materialize_18_cpw_size; i++) std::cout << cycles_per_warp_main_19_materialize_18[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_15);
hipFree(d_BUF_IDX_15);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_MAT18aggr0__tmp_attr0);
hipFree(d_MAT_IDX18);
free(MAT18aggr0__tmp_attr0);
}