#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_613318cdcb10(uint64_t* COUNT613318cf2450, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1993, Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT613318cf2450, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_613318cdcb10(uint64_t* BUF_613318cf2450, uint64_t* BUF_IDX_613318cf2450, HASHTABLE_INSERT HT_613318cf2450, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1993, Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_613318cf2450 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_613318cf2450 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_613318cf2450 = atomicAdd((int*)BUF_IDX_613318cf2450, 1);
HT_613318cf2450.insert(cuco::pair{KEY_613318cf2450, buf_idx_613318cf2450});
BUF_613318cf2450[buf_idx_613318cf2450 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_613318cdc830(uint64_t* BUF_613318cf2450, HASHTABLE_INSERT HT_613318cadbf0, HASHTABLE_PROBE HT_613318cf2450, DBI32Type* lineorder__lo_discount, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_discount = lineorder__lo_discount[tid];
if (!(evaluatePredicate(reg_lineorder__lo_discount, 1, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount, 3, Predicate::lte))) return;
auto reg_lineorder__lo_quantity = lineorder__lo_quantity[tid];
if (!(evaluatePredicate(reg_lineorder__lo_quantity, 25, Predicate::lt))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_613318cf2450 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_613318cf2450 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_613318cf2450.for_each(KEY_613318cf2450, [&] __device__ (auto const SLOT_613318cf2450) {

auto const [slot_first613318cf2450, slot_second613318cf2450] = SLOT_613318cf2450;
if (!(true)) return;
uint64_t KEY_613318cadbf0 = 0;
//Create aggregation hash table
HT_613318cadbf0.insert(cuco::pair{KEY_613318cadbf0, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_613318cdc830(uint64_t* BUF_613318cf2450, HASHTABLE_FIND HT_613318cadbf0, HASHTABLE_PROBE HT_613318cf2450, DBDecimalType* aggr0__tmp_attr0, DBI32Type* lineorder__lo_discount, DBDecimalType* lineorder__lo_extendedprice, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_discount = lineorder__lo_discount[tid];
if (!(evaluatePredicate(reg_lineorder__lo_discount, 1, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount, 3, Predicate::lte))) return;
auto reg_lineorder__lo_quantity = lineorder__lo_quantity[tid];
if (!(evaluatePredicate(reg_lineorder__lo_quantity, 25, Predicate::lt))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_613318cf2450 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_613318cf2450 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_613318cf2450.for_each(KEY_613318cf2450, [&] __device__ (auto const SLOT_613318cf2450) {
auto const [slot_first613318cf2450, slot_second613318cf2450] = SLOT_613318cf2450;
if (!(true)) return;
uint64_t KEY_613318cadbf0 = 0;
//Aggregate in hashtable
auto buf_idx_613318cadbf0 = HT_613318cadbf0.find(KEY_613318cadbf0)->second;
auto reg_lineorder__lo_extendedprice = lineorder__lo_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_extendedprice) * ((DBDecimalType)(reg_lineorder__lo_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_613318cadbf0], reg_map0__tmp_attr1);
});
}
__global__ void count_613318d04e30(uint64_t* COUNT613318c8b520, size_t COUNT613318cadbf0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT613318cadbf0) return;
//Materialize count
atomicAdd((int*)COUNT613318c8b520, 1);
}
__global__ void main_613318d04e30(size_t COUNT613318cadbf0, DBDecimalType* MAT613318c8b520aggr0__tmp_attr0, uint64_t* MAT_IDX613318c8b520, DBDecimalType* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT613318cadbf0) return;
//Materialize buffers
auto mat_idx613318c8b520 = atomicAdd((int*)MAT_IDX613318c8b520, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT613318c8b520aggr0__tmp_attr0[mat_idx613318c8b520] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT613318cf2450;
hipMalloc(&d_COUNT613318cf2450, sizeof(uint64_t));
hipMemset(d_COUNT613318cf2450, 0, sizeof(uint64_t));
count_613318cdcb10<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT613318cf2450, d_date__d_year, date_size);
uint64_t COUNT613318cf2450;
hipMemcpy(&COUNT613318cf2450, d_COUNT613318cf2450, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_613318cf2450;
hipMalloc(&d_BUF_IDX_613318cf2450, sizeof(uint64_t));
hipMemset(d_BUF_IDX_613318cf2450, 0, sizeof(uint64_t));
uint64_t* d_BUF_613318cf2450;
hipMalloc(&d_BUF_613318cf2450, sizeof(uint64_t) * COUNT613318cf2450 * 1);
auto d_HT_613318cf2450 = cuco::experimental::static_multimap{ (int)COUNT613318cf2450*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_613318cdcb10<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_613318cf2450, d_BUF_IDX_613318cf2450, d_HT_613318cf2450.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Create aggregation hash table
auto d_HT_613318cadbf0 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_613318cdc830<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_613318cf2450, d_HT_613318cadbf0.ref(cuco::insert), d_HT_613318cf2450.ref(cuco::for_each), d_lineorder__lo_discount, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
size_t COUNT613318cadbf0 = d_HT_613318cadbf0.size();
thrust::device_vector<int64_t> keys_613318cadbf0(COUNT613318cadbf0), vals_613318cadbf0(COUNT613318cadbf0);
d_HT_613318cadbf0.retrieve_all(keys_613318cadbf0.begin(), vals_613318cadbf0.begin());
d_HT_613318cadbf0.clear();
int64_t* raw_keys613318cadbf0 = thrust::raw_pointer_cast(keys_613318cadbf0.data());
insertKeys<<<std::ceil((float)COUNT613318cadbf0/128.), 128>>>(raw_keys613318cadbf0, d_HT_613318cadbf0.ref(cuco::insert), COUNT613318cadbf0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT613318cadbf0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT613318cadbf0);
main_613318cdc830<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_613318cf2450, d_HT_613318cadbf0.ref(cuco::find), d_HT_613318cf2450.ref(cuco::for_each), d_aggr0__tmp_attr0, d_lineorder__lo_discount, d_lineorder__lo_extendedprice, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
//Materialize count
uint64_t* d_COUNT613318c8b520;
hipMalloc(&d_COUNT613318c8b520, sizeof(uint64_t));
hipMemset(d_COUNT613318c8b520, 0, sizeof(uint64_t));
count_613318d04e30<<<std::ceil((float)COUNT613318cadbf0/128.), 128>>>(d_COUNT613318c8b520, COUNT613318cadbf0);
uint64_t COUNT613318c8b520;
hipMemcpy(&COUNT613318c8b520, d_COUNT613318c8b520, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX613318c8b520;
hipMalloc(&d_MAT_IDX613318c8b520, sizeof(uint64_t));
hipMemset(d_MAT_IDX613318c8b520, 0, sizeof(uint64_t));
auto MAT613318c8b520aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT613318c8b520);
DBDecimalType* d_MAT613318c8b520aggr0__tmp_attr0;
hipMalloc(&d_MAT613318c8b520aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT613318c8b520);
main_613318d04e30<<<std::ceil((float)COUNT613318cadbf0/128.), 128>>>(COUNT613318cadbf0, d_MAT613318c8b520aggr0__tmp_attr0, d_MAT_IDX613318c8b520, d_aggr0__tmp_attr0);
hipMemcpy(MAT613318c8b520aggr0__tmp_attr0, d_MAT613318c8b520aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT613318c8b520, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT613318c8b520; i++) { std::cout << "" << MAT613318c8b520aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_613318cf2450);
hipFree(d_BUF_IDX_613318cf2450);
hipFree(d_COUNT613318cf2450);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT613318c8b520);
hipFree(d_MAT613318c8b520aggr0__tmp_attr0);
hipFree(d_MAT_IDX613318c8b520);
free(MAT613318c8b520aggr0__tmp_attr0);
}