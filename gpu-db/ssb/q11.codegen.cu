#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5eadd96e7170(uint64_t* COUNT5eadd96fc3c0, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1993, Predicate::eq))) return;
//Materialize count
atomicAdd((int*)COUNT5eadd96fc3c0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5eadd96e7170(uint64_t* BUF_5eadd96fc3c0, uint64_t* BUF_IDX_5eadd96fc3c0, HASHTABLE_INSERT HT_5eadd96fc3c0, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1993, Predicate::eq))) return;
uint64_t KEY_5eadd96fc3c0 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5eadd96fc3c0 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_5eadd96fc3c0 = atomicAdd((int*)BUF_IDX_5eadd96fc3c0, 1);
HT_5eadd96fc3c0.insert(cuco::pair{KEY_5eadd96fc3c0, buf_idx_5eadd96fc3c0});
BUF_5eadd96fc3c0[buf_idx_5eadd96fc3c0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5eadd96e6e90(uint64_t* BUF_5eadd96fc3c0, HASHTABLE_INSERT HT_5eadd96b74d0, HASHTABLE_PROBE HT_5eadd96fc3c0, DBI32Type* lineorder__lo_discount, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {

uint64_t KEY_5eadd96b74d0 = 0;
//Create aggregation hash table
HT_5eadd96b74d0.insert(cuco::pair{KEY_5eadd96b74d0, 1});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5eadd96e6e90(uint64_t* BUF_5eadd96fc3c0, HASHTABLE_FIND HT_5eadd96b74d0, HASHTABLE_PROBE HT_5eadd96fc3c0, DBDecimalType* aggr0__tmp_attr0, DBI32Type* lineorder__lo_discount, DBDecimalType* lineorder__lo_extendedprice, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_discount = lineorder__lo_discount[tid];
__shared__ DBDecimalType agg;
__syncthreads();
agg = 0.0;
__syncthreads();
if ((evaluatePredicate(reg_lineorder__lo_discount, 1, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount, 3, Predicate::lte))) {
    auto reg_lineorder__lo_quantity = lineorder__lo_quantity[tid];
    if ((evaluatePredicate(reg_lineorder__lo_quantity, 25, Predicate::lt))) {
        uint64_t KEY_5eadd96fc3c0 = 0;
        auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];
        
        KEY_5eadd96fc3c0 |= reg_lineorder__lo_orderdate;
        //Probe Hash table
        HT_5eadd96fc3c0.for_each(KEY_5eadd96fc3c0, [&] __device__ (auto const SLOT_5eadd96fc3c0) {
        auto const [slot_first5eadd96fc3c0, slot_second5eadd96fc3c0] = SLOT_5eadd96fc3c0;
        if (!(true)) return;
        uint64_t KEY_5eadd96b74d0 = 0;
        //Aggregate in hashtable
        auto buf_idx_5eadd96b74d0 = HT_5eadd96b74d0.find(KEY_5eadd96b74d0)->second;
        auto reg_lineorder__lo_extendedprice = lineorder__lo_extendedprice[tid];
        auto reg_map0__tmp_attr1 = (reg_lineorder__lo_extendedprice) * ((DBDecimalType)(reg_lineorder__lo_discount));
        aggregate_sum(&agg, reg_map0__tmp_attr1);
        });
    }
}
__syncthreads();
if (threadIdx.x == 0) {
    aggregate_sum(&aggr0__tmp_attr0[0], agg);
}
}
__global__ void count_5eadd970db50(uint64_t* COUNT5eadd9696480, size_t COUNT5eadd96b74d0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5eadd96b74d0) return;
//Materialize count
atomicAdd((int*)COUNT5eadd9696480, 1);
}
__global__ void main_5eadd970db50(size_t COUNT5eadd96b74d0, DBDecimalType* MAT5eadd9696480aggr0__tmp_attr0, uint64_t* MAT_IDX5eadd9696480, DBDecimalType* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5eadd96b74d0) return;
//Materialize buffers
auto mat_idx5eadd9696480 = atomicAdd((int*)MAT_IDX5eadd9696480, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5eadd9696480aggr0__tmp_attr0[mat_idx5eadd9696480] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT5eadd96fc3c0;
hipMalloc(&d_COUNT5eadd96fc3c0, sizeof(uint64_t));
hipMemset(d_COUNT5eadd96fc3c0, 0, sizeof(uint64_t));
count_5eadd96e7170<<<std::ceil((float)date_size/32.), 32>>>(d_COUNT5eadd96fc3c0, d_date__d_year, date_size);
uint64_t COUNT5eadd96fc3c0;
hipMemcpy(&COUNT5eadd96fc3c0, d_COUNT5eadd96fc3c0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5eadd96fc3c0;
hipMalloc(&d_BUF_IDX_5eadd96fc3c0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5eadd96fc3c0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5eadd96fc3c0;
hipMalloc(&d_BUF_5eadd96fc3c0, sizeof(uint64_t) * COUNT5eadd96fc3c0 * 1);
auto d_HT_5eadd96fc3c0 = cuco::experimental::static_multimap{ (int)COUNT5eadd96fc3c0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5eadd96e7170<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_5eadd96fc3c0, d_BUF_IDX_5eadd96fc3c0, d_HT_5eadd96fc3c0.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Create aggregation hash table
auto d_HT_5eadd96b74d0 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5eadd96e6e90<<<1, 1>>>(d_BUF_5eadd96fc3c0, d_HT_5eadd96b74d0.ref(cuco::insert), d_HT_5eadd96fc3c0.ref(cuco::for_each), d_lineorder__lo_discount, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
size_t COUNT5eadd96b74d0 = d_HT_5eadd96b74d0.size();
thrust::device_vector<int64_t> keys_5eadd96b74d0(COUNT5eadd96b74d0), vals_5eadd96b74d0(COUNT5eadd96b74d0);
d_HT_5eadd96b74d0.retrieve_all(keys_5eadd96b74d0.begin(), vals_5eadd96b74d0.begin());
d_HT_5eadd96b74d0.clear();
int64_t* raw_keys5eadd96b74d0 = thrust::raw_pointer_cast(keys_5eadd96b74d0.data());
insertKeys<<<std::ceil((float)COUNT5eadd96b74d0/32.), 32>>>(raw_keys5eadd96b74d0, d_HT_5eadd96b74d0.ref(cuco::insert), COUNT5eadd96b74d0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5eadd96b74d0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5eadd96b74d0);
main_5eadd96e6e90<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5eadd96fc3c0, d_HT_5eadd96b74d0.ref(cuco::find), d_HT_5eadd96fc3c0.ref(cuco::for_each), d_aggr0__tmp_attr0, d_lineorder__lo_discount, d_lineorder__lo_extendedprice, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
//Materialize count
uint64_t* d_COUNT5eadd9696480;
hipMalloc(&d_COUNT5eadd9696480, sizeof(uint64_t));
hipMemset(d_COUNT5eadd9696480, 0, sizeof(uint64_t));
count_5eadd970db50<<<std::ceil((float)COUNT5eadd96b74d0/32.), 32>>>(d_COUNT5eadd9696480, COUNT5eadd96b74d0);
uint64_t COUNT5eadd9696480;
hipMemcpy(&COUNT5eadd9696480, d_COUNT5eadd9696480, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5eadd9696480;
hipMalloc(&d_MAT_IDX5eadd9696480, sizeof(uint64_t));
hipMemset(d_MAT_IDX5eadd9696480, 0, sizeof(uint64_t));
auto MAT5eadd9696480aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5eadd9696480);
DBDecimalType* d_MAT5eadd9696480aggr0__tmp_attr0;
hipMalloc(&d_MAT5eadd9696480aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5eadd9696480);
main_5eadd970db50<<<std::ceil((float)COUNT5eadd96b74d0/32.), 32>>>(COUNT5eadd96b74d0, d_MAT5eadd9696480aggr0__tmp_attr0, d_MAT_IDX5eadd9696480, d_aggr0__tmp_attr0);
hipMemcpy(MAT5eadd9696480aggr0__tmp_attr0, d_MAT5eadd9696480aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5eadd9696480, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5eadd9696480; i++) { std::cout << MAT5eadd9696480aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_5eadd96fc3c0);
hipFree(d_BUF_IDX_5eadd96fc3c0);
hipFree(d_COUNT5eadd96fc3c0);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5eadd9696480);
hipFree(d_MAT5eadd9696480aggr0__tmp_attr0);
hipFree(d_MAT_IDX5eadd9696480);
free(MAT5eadd9696480aggr0__tmp_attr0);
}