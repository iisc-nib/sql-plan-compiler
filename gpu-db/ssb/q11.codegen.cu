#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_64e1ea15f8d0(uint64_t* COUNT64e1ea174ab0, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1993, Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT64e1ea174ab0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_64e1ea15f8d0(uint64_t* BUF_64e1ea174ab0, uint64_t* BUF_IDX_64e1ea174ab0, HASHTABLE_INSERT HT_64e1ea174ab0, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1993, Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_64e1ea174ab0 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_64e1ea174ab0 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_64e1ea174ab0 = atomicAdd((int*)BUF_IDX_64e1ea174ab0, 1);
HT_64e1ea174ab0.insert(cuco::pair{KEY_64e1ea174ab0, buf_idx_64e1ea174ab0});
BUF_64e1ea174ab0[buf_idx_64e1ea174ab0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_64e1ea15f5c0(uint64_t* BUF_64e1ea174ab0, HASHTABLE_INSERT HT_64e1ea130550, HASHTABLE_PROBE HT_64e1ea174ab0, DBI32Type* lineorder__lo_discount, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_discount = lineorder__lo_discount[tid];
if (!(evaluatePredicate(reg_lineorder__lo_discount, 1, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount, 3, Predicate::lte))) return;
auto reg_lineorder__lo_quantity = lineorder__lo_quantity[tid];
if (!(evaluatePredicate(reg_lineorder__lo_quantity, 25, Predicate::lt))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_64e1ea174ab0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_64e1ea174ab0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_64e1ea174ab0.for_each(KEY_64e1ea174ab0, [&] __device__ (auto const SLOT_64e1ea174ab0) {

auto const [slot_first64e1ea174ab0, slot_second64e1ea174ab0] = SLOT_64e1ea174ab0;
if (!(true)) return;
uint64_t KEY_64e1ea130550 = 0;
//Create aggregation hash table
HT_64e1ea130550.insert(cuco::pair{KEY_64e1ea130550, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_64e1ea15f5c0(uint64_t* BUF_64e1ea174ab0, HASHTABLE_FIND HT_64e1ea130550, HASHTABLE_PROBE HT_64e1ea174ab0, DBDecimalType* aggr0__tmp_attr0, DBI32Type* lineorder__lo_discount, DBDecimalType* lineorder__lo_extendedprice, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_discount = lineorder__lo_discount[tid];
if (!(evaluatePredicate(reg_lineorder__lo_discount, 1, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount, 3, Predicate::lte))) return;
auto reg_lineorder__lo_quantity = lineorder__lo_quantity[tid];
if (!(evaluatePredicate(reg_lineorder__lo_quantity, 25, Predicate::lt))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_64e1ea174ab0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_64e1ea174ab0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_64e1ea174ab0.for_each(KEY_64e1ea174ab0, [&] __device__ (auto const SLOT_64e1ea174ab0) {
auto const [slot_first64e1ea174ab0, slot_second64e1ea174ab0] = SLOT_64e1ea174ab0;
if (!(true)) return;
uint64_t KEY_64e1ea130550 = 0;
//Aggregate in hashtable
auto buf_idx_64e1ea130550 = HT_64e1ea130550.find(KEY_64e1ea130550)->second;
auto reg_lineorder__lo_extendedprice = lineorder__lo_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_extendedprice) * ((DBDecimalType)(reg_lineorder__lo_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_64e1ea130550], reg_map0__tmp_attr1);
});
}
__global__ void count_64e1ea187b80(uint64_t* COUNT64e1ea10e7b0, size_t COUNT64e1ea130550) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT64e1ea130550) return;
//Materialize count
atomicAdd((int*)COUNT64e1ea10e7b0, 1);
}
__global__ void main_64e1ea187b80(size_t COUNT64e1ea130550, DBDecimalType* MAT64e1ea10e7b0aggr0__tmp_attr0, uint64_t* MAT_IDX64e1ea10e7b0, DBDecimalType* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT64e1ea130550) return;
//Materialize buffers
auto mat_idx64e1ea10e7b0 = atomicAdd((int*)MAT_IDX64e1ea10e7b0, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT64e1ea10e7b0aggr0__tmp_attr0[mat_idx64e1ea10e7b0] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map) {
//Materialize count
uint64_t* d_COUNT64e1ea174ab0;
hipMalloc(&d_COUNT64e1ea174ab0, sizeof(uint64_t));
hipMemset(d_COUNT64e1ea174ab0, 0, sizeof(uint64_t));
count_64e1ea15f8d0<<<std::ceil((float)date_size/32.), 32>>>(d_COUNT64e1ea174ab0, d_date__d_year, date_size);
uint64_t COUNT64e1ea174ab0;
hipMemcpy(&COUNT64e1ea174ab0, d_COUNT64e1ea174ab0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_64e1ea174ab0;
hipMalloc(&d_BUF_IDX_64e1ea174ab0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_64e1ea174ab0, 0, sizeof(uint64_t));
uint64_t* d_BUF_64e1ea174ab0;
hipMalloc(&d_BUF_64e1ea174ab0, sizeof(uint64_t) * COUNT64e1ea174ab0 * 1);
auto d_HT_64e1ea174ab0 = cuco::experimental::static_multimap{ (int)COUNT64e1ea174ab0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_64e1ea15f8d0<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_64e1ea174ab0, d_BUF_IDX_64e1ea174ab0, d_HT_64e1ea174ab0.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Create aggregation hash table
auto d_HT_64e1ea130550 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_64e1ea15f5c0<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_64e1ea174ab0, d_HT_64e1ea130550.ref(cuco::insert), d_HT_64e1ea174ab0.ref(cuco::for_each), d_lineorder__lo_discount, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
size_t COUNT64e1ea130550 = d_HT_64e1ea130550.size();
thrust::device_vector<int64_t> keys_64e1ea130550(COUNT64e1ea130550), vals_64e1ea130550(COUNT64e1ea130550);
d_HT_64e1ea130550.retrieve_all(keys_64e1ea130550.begin(), vals_64e1ea130550.begin());
d_HT_64e1ea130550.clear();
int64_t* raw_keys64e1ea130550 = thrust::raw_pointer_cast(keys_64e1ea130550.data());
insertKeys<<<std::ceil((float)COUNT64e1ea130550/32.), 32>>>(raw_keys64e1ea130550, d_HT_64e1ea130550.ref(cuco::insert), COUNT64e1ea130550);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT64e1ea130550);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT64e1ea130550);
main_64e1ea15f5c0<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_64e1ea174ab0, d_HT_64e1ea130550.ref(cuco::find), d_HT_64e1ea174ab0.ref(cuco::for_each), d_aggr0__tmp_attr0, d_lineorder__lo_discount, d_lineorder__lo_extendedprice, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
//Materialize count
uint64_t* d_COUNT64e1ea10e7b0;
hipMalloc(&d_COUNT64e1ea10e7b0, sizeof(uint64_t));
hipMemset(d_COUNT64e1ea10e7b0, 0, sizeof(uint64_t));
count_64e1ea187b80<<<std::ceil((float)COUNT64e1ea130550/32.), 32>>>(d_COUNT64e1ea10e7b0, COUNT64e1ea130550);
uint64_t COUNT64e1ea10e7b0;
hipMemcpy(&COUNT64e1ea10e7b0, d_COUNT64e1ea10e7b0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX64e1ea10e7b0;
hipMalloc(&d_MAT_IDX64e1ea10e7b0, sizeof(uint64_t));
hipMemset(d_MAT_IDX64e1ea10e7b0, 0, sizeof(uint64_t));
auto MAT64e1ea10e7b0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT64e1ea10e7b0);
DBDecimalType* d_MAT64e1ea10e7b0aggr0__tmp_attr0;
hipMalloc(&d_MAT64e1ea10e7b0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT64e1ea10e7b0);
main_64e1ea187b80<<<std::ceil((float)COUNT64e1ea130550/32.), 32>>>(COUNT64e1ea130550, d_MAT64e1ea10e7b0aggr0__tmp_attr0, d_MAT_IDX64e1ea10e7b0, d_aggr0__tmp_attr0);
hipMemcpy(MAT64e1ea10e7b0aggr0__tmp_attr0, d_MAT64e1ea10e7b0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT64e1ea10e7b0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT64e1ea10e7b0; i++) { std::cout << MAT64e1ea10e7b0aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_64e1ea174ab0);
hipFree(d_BUF_IDX_64e1ea174ab0);
hipFree(d_COUNT64e1ea174ab0);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT64e1ea10e7b0);
hipFree(d_MAT64e1ea10e7b0aggr0__tmp_attr0);
hipFree(d_MAT_IDX64e1ea10e7b0);
free(MAT64e1ea10e7b0aggr0__tmp_attr0);
}