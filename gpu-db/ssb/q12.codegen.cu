#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5fc88e7bba30(uint64_t* COUNT5fc88e7d0c10, DBI32Type* date__d_yearmonthnum, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_yearmonthnum = date__d_yearmonthnum[tid];
if (!(evaluatePredicate(reg_date__d_yearmonthnum, 199401, Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5fc88e7d0c10, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5fc88e7bba30(uint64_t* BUF_5fc88e7d0c10, uint64_t* BUF_IDX_5fc88e7d0c10, HASHTABLE_INSERT HT_5fc88e7d0c10, DBI32Type* date__d_datekey, DBI32Type* date__d_yearmonthnum, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_yearmonthnum = date__d_yearmonthnum[tid];
if (!(evaluatePredicate(reg_date__d_yearmonthnum, 199401, Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5fc88e7d0c10 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5fc88e7d0c10 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_5fc88e7d0c10 = atomicAdd((int*)BUF_IDX_5fc88e7d0c10, 1);
HT_5fc88e7d0c10.insert(cuco::pair{KEY_5fc88e7d0c10, buf_idx_5fc88e7d0c10});
BUF_5fc88e7d0c10[buf_idx_5fc88e7d0c10 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5fc88e7bb580(uint64_t* BUF_5fc88e7d0c10, HASHTABLE_INSERT HT_5fc88e78c8e0, HASHTABLE_PROBE HT_5fc88e7d0c10, DBI32Type* lineorder__lo_discount, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_discount = lineorder__lo_discount[tid];
if (!(evaluatePredicate(reg_lineorder__lo_discount, 4, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount, 6, Predicate::lte))) return;
auto reg_lineorder__lo_quantity = lineorder__lo_quantity[tid];
if (!(evaluatePredicate(reg_lineorder__lo_quantity, 26, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_quantity, 35, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5fc88e7d0c10 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5fc88e7d0c10 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5fc88e7d0c10.for_each(KEY_5fc88e7d0c10, [&] __device__ (auto const SLOT_5fc88e7d0c10) {

auto const [slot_first5fc88e7d0c10, slot_second5fc88e7d0c10] = SLOT_5fc88e7d0c10;
if (!(true)) return;
uint64_t KEY_5fc88e78c8e0 = 0;
//Create aggregation hash table
HT_5fc88e78c8e0.insert(cuco::pair{KEY_5fc88e78c8e0, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5fc88e7bb580(uint64_t* BUF_5fc88e7d0c10, HASHTABLE_FIND HT_5fc88e78c8e0, HASHTABLE_PROBE HT_5fc88e7d0c10, DBDecimalType* aggr0__tmp_attr0, DBI32Type* lineorder__lo_discount, DBDecimalType* lineorder__lo_extendedprice, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_discount = lineorder__lo_discount[tid];
if (!(evaluatePredicate(reg_lineorder__lo_discount, 4, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount, 6, Predicate::lte))) return;
auto reg_lineorder__lo_quantity = lineorder__lo_quantity[tid];
if (!(evaluatePredicate(reg_lineorder__lo_quantity, 26, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_quantity, 35, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5fc88e7d0c10 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5fc88e7d0c10 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5fc88e7d0c10.for_each(KEY_5fc88e7d0c10, [&] __device__ (auto const SLOT_5fc88e7d0c10) {
auto const [slot_first5fc88e7d0c10, slot_second5fc88e7d0c10] = SLOT_5fc88e7d0c10;
if (!(true)) return;
uint64_t KEY_5fc88e78c8e0 = 0;
//Aggregate in hashtable
auto buf_idx_5fc88e78c8e0 = HT_5fc88e78c8e0.find(KEY_5fc88e78c8e0)->second;
auto reg_lineorder__lo_extendedprice = lineorder__lo_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_extendedprice) * ((DBDecimalType)(reg_lineorder__lo_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5fc88e78c8e0], reg_map0__tmp_attr1);
});
}
__global__ void count_5fc88e7e3c50(size_t COUNT5fc88e78c8e0, uint64_t* COUNT5fc88e79ec80) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5fc88e78c8e0) return;
//Materialize count
atomicAdd((int*)COUNT5fc88e79ec80, 1);
}
__global__ void main_5fc88e7e3c50(size_t COUNT5fc88e78c8e0, DBDecimalType* MAT5fc88e79ec80aggr0__tmp_attr0, uint64_t* MAT_IDX5fc88e79ec80, DBDecimalType* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5fc88e78c8e0) return;
//Materialize buffers
auto mat_idx5fc88e79ec80 = atomicAdd((int*)MAT_IDX5fc88e79ec80, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5fc88e79ec80aggr0__tmp_attr0[mat_idx5fc88e79ec80] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map) {
//Materialize count
uint64_t* d_COUNT5fc88e7d0c10;
hipMalloc(&d_COUNT5fc88e7d0c10, sizeof(uint64_t));
hipMemset(d_COUNT5fc88e7d0c10, 0, sizeof(uint64_t));
count_5fc88e7bba30<<<std::ceil((float)date_size/32.), 32>>>(d_COUNT5fc88e7d0c10, d_date__d_yearmonthnum, date_size);
uint64_t COUNT5fc88e7d0c10;
hipMemcpy(&COUNT5fc88e7d0c10, d_COUNT5fc88e7d0c10, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5fc88e7d0c10;
hipMalloc(&d_BUF_IDX_5fc88e7d0c10, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5fc88e7d0c10, 0, sizeof(uint64_t));
uint64_t* d_BUF_5fc88e7d0c10;
hipMalloc(&d_BUF_5fc88e7d0c10, sizeof(uint64_t) * COUNT5fc88e7d0c10 * 1);
auto d_HT_5fc88e7d0c10 = cuco::experimental::static_multimap{ (int)COUNT5fc88e7d0c10*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5fc88e7bba30<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_5fc88e7d0c10, d_BUF_IDX_5fc88e7d0c10, d_HT_5fc88e7d0c10.ref(cuco::insert), d_date__d_datekey, d_date__d_yearmonthnum, date_size);
//Create aggregation hash table
auto d_HT_5fc88e78c8e0 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5fc88e7bb580<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_5fc88e7d0c10, d_HT_5fc88e78c8e0.ref(cuco::insert), d_HT_5fc88e7d0c10.ref(cuco::for_each), d_lineorder__lo_discount, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
size_t COUNT5fc88e78c8e0 = d_HT_5fc88e78c8e0.size();
thrust::device_vector<int64_t> keys_5fc88e78c8e0(COUNT5fc88e78c8e0), vals_5fc88e78c8e0(COUNT5fc88e78c8e0);
d_HT_5fc88e78c8e0.retrieve_all(keys_5fc88e78c8e0.begin(), vals_5fc88e78c8e0.begin());
d_HT_5fc88e78c8e0.clear();
int64_t* raw_keys5fc88e78c8e0 = thrust::raw_pointer_cast(keys_5fc88e78c8e0.data());
insertKeys<<<std::ceil((float)COUNT5fc88e78c8e0/32.), 32>>>(raw_keys5fc88e78c8e0, d_HT_5fc88e78c8e0.ref(cuco::insert), COUNT5fc88e78c8e0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5fc88e78c8e0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5fc88e78c8e0);
main_5fc88e7bb580<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_5fc88e7d0c10, d_HT_5fc88e78c8e0.ref(cuco::find), d_HT_5fc88e7d0c10.ref(cuco::for_each), d_aggr0__tmp_attr0, d_lineorder__lo_discount, d_lineorder__lo_extendedprice, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
//Materialize count
uint64_t* d_COUNT5fc88e79ec80;
hipMalloc(&d_COUNT5fc88e79ec80, sizeof(uint64_t));
hipMemset(d_COUNT5fc88e79ec80, 0, sizeof(uint64_t));
count_5fc88e7e3c50<<<std::ceil((float)COUNT5fc88e78c8e0/32.), 32>>>(COUNT5fc88e78c8e0, d_COUNT5fc88e79ec80);
uint64_t COUNT5fc88e79ec80;
hipMemcpy(&COUNT5fc88e79ec80, d_COUNT5fc88e79ec80, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5fc88e79ec80;
hipMalloc(&d_MAT_IDX5fc88e79ec80, sizeof(uint64_t));
hipMemset(d_MAT_IDX5fc88e79ec80, 0, sizeof(uint64_t));
auto MAT5fc88e79ec80aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5fc88e79ec80);
DBDecimalType* d_MAT5fc88e79ec80aggr0__tmp_attr0;
hipMalloc(&d_MAT5fc88e79ec80aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5fc88e79ec80);
main_5fc88e7e3c50<<<std::ceil((float)COUNT5fc88e78c8e0/32.), 32>>>(COUNT5fc88e78c8e0, d_MAT5fc88e79ec80aggr0__tmp_attr0, d_MAT_IDX5fc88e79ec80, d_aggr0__tmp_attr0);
hipMemcpy(MAT5fc88e79ec80aggr0__tmp_attr0, d_MAT5fc88e79ec80aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5fc88e79ec80, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5fc88e79ec80; i++) { std::cout << MAT5fc88e79ec80aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_5fc88e7d0c10);
hipFree(d_BUF_IDX_5fc88e7d0c10);
hipFree(d_COUNT5fc88e7d0c10);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5fc88e79ec80);
hipFree(d_MAT5fc88e79ec80aggr0__tmp_attr0);
hipFree(d_MAT_IDX5fc88e79ec80);
free(MAT5fc88e79ec80aggr0__tmp_attr0);
}