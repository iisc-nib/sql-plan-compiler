#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_626db78aa210(uint64_t* COUNT626db78bf120, DBI32Type* date__d_yearmonthnum, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_yearmonthnum = date__d_yearmonthnum[tid];
if (!(evaluatePredicate(reg_date__d_yearmonthnum, 199401, Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT626db78bf120, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_626db78aa210(uint64_t* BUF_626db78bf120, uint64_t* BUF_IDX_626db78bf120, HASHTABLE_INSERT HT_626db78bf120, DBI32Type* date__d_datekey, DBI32Type* date__d_yearmonthnum, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_yearmonthnum = date__d_yearmonthnum[tid];
if (!(evaluatePredicate(reg_date__d_yearmonthnum, 199401, Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_626db78bf120 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_626db78bf120 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_626db78bf120 = atomicAdd((int*)BUF_IDX_626db78bf120, 1);
HT_626db78bf120.insert(cuco::pair{KEY_626db78bf120, buf_idx_626db78bf120});
BUF_626db78bf120[buf_idx_626db78bf120 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_626db78a9f10(uint64_t* BUF_626db78bf120, HASHTABLE_INSERT HT_626db787ae30, HASHTABLE_PROBE HT_626db78bf120, DBI32Type* lineorder__lo_discount, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_discount = lineorder__lo_discount[tid];
if (!(evaluatePredicate(reg_lineorder__lo_discount, 4, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount, 6, Predicate::lte))) return;
auto reg_lineorder__lo_quantity = lineorder__lo_quantity[tid];
if (!(evaluatePredicate(reg_lineorder__lo_quantity, 26, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_quantity, 35, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_626db78bf120 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_626db78bf120 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_626db78bf120.for_each(KEY_626db78bf120, [&] __device__ (auto const SLOT_626db78bf120) {

auto const [slot_first626db78bf120, slot_second626db78bf120] = SLOT_626db78bf120;
if (!(true)) return;
uint64_t KEY_626db787ae30 = 0;
//Create aggregation hash table
HT_626db787ae30.insert(cuco::pair{KEY_626db787ae30, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_626db78a9f10(uint64_t* BUF_626db78bf120, HASHTABLE_FIND HT_626db787ae30, HASHTABLE_PROBE HT_626db78bf120, DBDecimalType* aggr0__tmp_attr0, DBI32Type* lineorder__lo_discount, DBDecimalType* lineorder__lo_extendedprice, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_discount = lineorder__lo_discount[tid];
if (!(evaluatePredicate(reg_lineorder__lo_discount, 4, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount, 6, Predicate::lte))) return;
auto reg_lineorder__lo_quantity = lineorder__lo_quantity[tid];
if (!(evaluatePredicate(reg_lineorder__lo_quantity, 26, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_quantity, 35, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_626db78bf120 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_626db78bf120 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_626db78bf120.for_each(KEY_626db78bf120, [&] __device__ (auto const SLOT_626db78bf120) {
auto const [slot_first626db78bf120, slot_second626db78bf120] = SLOT_626db78bf120;
if (!(true)) return;
uint64_t KEY_626db787ae30 = 0;
//Aggregate in hashtable
auto buf_idx_626db787ae30 = HT_626db787ae30.find(KEY_626db787ae30)->second;
auto reg_lineorder__lo_extendedprice = lineorder__lo_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_extendedprice) * ((DBDecimalType)(reg_lineorder__lo_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_626db787ae30], reg_map0__tmp_attr1);
});
}
__global__ void count_626db78d2eb0(size_t COUNT626db787ae30, uint64_t* COUNT626db788d610) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT626db787ae30) return;
//Materialize count
atomicAdd((int*)COUNT626db788d610, 1);
}
__global__ void main_626db78d2eb0(size_t COUNT626db787ae30, DBDecimalType* MAT626db788d610aggr0__tmp_attr0, uint64_t* MAT_IDX626db788d610, DBDecimalType* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT626db787ae30) return;
//Materialize buffers
auto mat_idx626db788d610 = atomicAdd((int*)MAT_IDX626db788d610, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT626db788d610aggr0__tmp_attr0[mat_idx626db788d610] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT626db78bf120;
hipMalloc(&d_COUNT626db78bf120, sizeof(uint64_t));
hipMemset(d_COUNT626db78bf120, 0, sizeof(uint64_t));
count_626db78aa210<<<std::ceil((float)date_size/32.), 32>>>(d_COUNT626db78bf120, d_date__d_yearmonthnum, date_size);
uint64_t COUNT626db78bf120;
hipMemcpy(&COUNT626db78bf120, d_COUNT626db78bf120, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_626db78bf120;
hipMalloc(&d_BUF_IDX_626db78bf120, sizeof(uint64_t));
hipMemset(d_BUF_IDX_626db78bf120, 0, sizeof(uint64_t));
uint64_t* d_BUF_626db78bf120;
hipMalloc(&d_BUF_626db78bf120, sizeof(uint64_t) * COUNT626db78bf120 * 1);
auto d_HT_626db78bf120 = cuco::experimental::static_multimap{ (int)COUNT626db78bf120*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_626db78aa210<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_626db78bf120, d_BUF_IDX_626db78bf120, d_HT_626db78bf120.ref(cuco::insert), d_date__d_datekey, d_date__d_yearmonthnum, date_size);
//Create aggregation hash table
auto d_HT_626db787ae30 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_626db78a9f10<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_626db78bf120, d_HT_626db787ae30.ref(cuco::insert), d_HT_626db78bf120.ref(cuco::for_each), d_lineorder__lo_discount, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
size_t COUNT626db787ae30 = d_HT_626db787ae30.size();
thrust::device_vector<int64_t> keys_626db787ae30(COUNT626db787ae30), vals_626db787ae30(COUNT626db787ae30);
d_HT_626db787ae30.retrieve_all(keys_626db787ae30.begin(), vals_626db787ae30.begin());
d_HT_626db787ae30.clear();
int64_t* raw_keys626db787ae30 = thrust::raw_pointer_cast(keys_626db787ae30.data());
insertKeys<<<std::ceil((float)COUNT626db787ae30/32.), 32>>>(raw_keys626db787ae30, d_HT_626db787ae30.ref(cuco::insert), COUNT626db787ae30);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT626db787ae30);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT626db787ae30);
main_626db78a9f10<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_626db78bf120, d_HT_626db787ae30.ref(cuco::find), d_HT_626db78bf120.ref(cuco::for_each), d_aggr0__tmp_attr0, d_lineorder__lo_discount, d_lineorder__lo_extendedprice, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
//Materialize count
uint64_t* d_COUNT626db788d610;
hipMalloc(&d_COUNT626db788d610, sizeof(uint64_t));
hipMemset(d_COUNT626db788d610, 0, sizeof(uint64_t));
count_626db78d2eb0<<<std::ceil((float)COUNT626db787ae30/32.), 32>>>(COUNT626db787ae30, d_COUNT626db788d610);
uint64_t COUNT626db788d610;
hipMemcpy(&COUNT626db788d610, d_COUNT626db788d610, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX626db788d610;
hipMalloc(&d_MAT_IDX626db788d610, sizeof(uint64_t));
hipMemset(d_MAT_IDX626db788d610, 0, sizeof(uint64_t));
auto MAT626db788d610aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT626db788d610);
DBDecimalType* d_MAT626db788d610aggr0__tmp_attr0;
hipMalloc(&d_MAT626db788d610aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT626db788d610);
main_626db78d2eb0<<<std::ceil((float)COUNT626db787ae30/32.), 32>>>(COUNT626db787ae30, d_MAT626db788d610aggr0__tmp_attr0, d_MAT_IDX626db788d610, d_aggr0__tmp_attr0);
hipMemcpy(MAT626db788d610aggr0__tmp_attr0, d_MAT626db788d610aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT626db788d610, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT626db788d610; i++) { std::cout << MAT626db788d610aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_626db78bf120);
hipFree(d_BUF_IDX_626db78bf120);
hipFree(d_COUNT626db78bf120);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT626db788d610);
hipFree(d_MAT626db788d610aggr0__tmp_attr0);
hipFree(d_MAT_IDX626db788d610);
free(MAT626db788d610aggr0__tmp_attr0);
}