#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_634ccc6701a0(uint64_t* COUNT634ccc688bb0, DBI32Type* date__d_yearmonthnum, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_yearmonthnum = date__d_yearmonthnum[tid];
if (!(evaluatePredicate(reg_date__d_yearmonthnum, 199401, Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT634ccc688bb0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_634ccc6701a0(uint64_t* BUF_634ccc688bb0, uint64_t* BUF_IDX_634ccc688bb0, HASHTABLE_INSERT HT_634ccc688bb0, DBI32Type* date__d_datekey, DBI32Type* date__d_yearmonthnum, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_yearmonthnum = date__d_yearmonthnum[tid];
if (!(evaluatePredicate(reg_date__d_yearmonthnum, 199401, Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_634ccc688bb0 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_634ccc688bb0 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_634ccc688bb0 = atomicAdd((int*)BUF_IDX_634ccc688bb0, 1);
HT_634ccc688bb0.insert(cuco::pair{KEY_634ccc688bb0, buf_idx_634ccc688bb0});
BUF_634ccc688bb0[buf_idx_634ccc688bb0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_634ccc66fcf0(uint64_t* BUF_634ccc688bb0, HASHTABLE_INSERT HT_634ccc640bf0, HASHTABLE_PROBE HT_634ccc688bb0, DBI32Type* lineorder__lo_discount, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_discount = lineorder__lo_discount[tid];
if (!(evaluatePredicate(reg_lineorder__lo_discount, 4, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount, 6, Predicate::lte))) return;
auto reg_lineorder__lo_quantity = lineorder__lo_quantity[tid];
if (!(evaluatePredicate(reg_lineorder__lo_quantity, 26, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_quantity, 35, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_634ccc688bb0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_634ccc688bb0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_634ccc688bb0.for_each(KEY_634ccc688bb0, [&] __device__ (auto const SLOT_634ccc688bb0) {

auto const [slot_first634ccc688bb0, slot_second634ccc688bb0] = SLOT_634ccc688bb0;
if (!(true)) return;
uint64_t KEY_634ccc640bf0 = 0;
//Create aggregation hash table
HT_634ccc640bf0.insert(cuco::pair{KEY_634ccc640bf0, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_634ccc66fcf0(uint64_t* BUF_634ccc688bb0, HASHTABLE_FIND HT_634ccc640bf0, HASHTABLE_PROBE HT_634ccc688bb0, DBDecimalType* aggr0__tmp_attr0, DBI32Type* lineorder__lo_discount, DBDecimalType* lineorder__lo_extendedprice, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_discount = lineorder__lo_discount[tid];
if (!(evaluatePredicate(reg_lineorder__lo_discount, 4, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount, 6, Predicate::lte))) return;
auto reg_lineorder__lo_quantity = lineorder__lo_quantity[tid];
if (!(evaluatePredicate(reg_lineorder__lo_quantity, 26, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_quantity, 35, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_634ccc688bb0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_634ccc688bb0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_634ccc688bb0.for_each(KEY_634ccc688bb0, [&] __device__ (auto const SLOT_634ccc688bb0) {
auto const [slot_first634ccc688bb0, slot_second634ccc688bb0] = SLOT_634ccc688bb0;
if (!(true)) return;
uint64_t KEY_634ccc640bf0 = 0;
//Aggregate in hashtable
auto buf_idx_634ccc640bf0 = HT_634ccc640bf0.find(KEY_634ccc640bf0)->second;
auto reg_lineorder__lo_extendedprice = lineorder__lo_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_extendedprice) * ((DBDecimalType)(reg_lineorder__lo_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_634ccc640bf0], reg_map0__tmp_attr1);
});
}
__global__ void count_634ccc698fe0(size_t COUNT634ccc640bf0, uint64_t* COUNT634ccc652f60) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT634ccc640bf0) return;
//Materialize count
atomicAdd((int*)COUNT634ccc652f60, 1);
}
__global__ void main_634ccc698fe0(size_t COUNT634ccc640bf0, DBDecimalType* MAT634ccc652f60aggr0__tmp_attr0, uint64_t* MAT_IDX634ccc652f60, DBDecimalType* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT634ccc640bf0) return;
//Materialize buffers
auto mat_idx634ccc652f60 = atomicAdd((int*)MAT_IDX634ccc652f60, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT634ccc652f60aggr0__tmp_attr0[mat_idx634ccc652f60] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT634ccc688bb0;
hipMalloc(&d_COUNT634ccc688bb0, sizeof(uint64_t));
hipMemset(d_COUNT634ccc688bb0, 0, sizeof(uint64_t));
count_634ccc6701a0<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT634ccc688bb0, d_date__d_yearmonthnum, date_size);
uint64_t COUNT634ccc688bb0;
hipMemcpy(&COUNT634ccc688bb0, d_COUNT634ccc688bb0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_634ccc688bb0;
hipMalloc(&d_BUF_IDX_634ccc688bb0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_634ccc688bb0, 0, sizeof(uint64_t));
uint64_t* d_BUF_634ccc688bb0;
hipMalloc(&d_BUF_634ccc688bb0, sizeof(uint64_t) * COUNT634ccc688bb0 * 1);
auto d_HT_634ccc688bb0 = cuco::experimental::static_multimap{ (int)COUNT634ccc688bb0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_634ccc6701a0<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_634ccc688bb0, d_BUF_IDX_634ccc688bb0, d_HT_634ccc688bb0.ref(cuco::insert), d_date__d_datekey, d_date__d_yearmonthnum, date_size);
//Create aggregation hash table
auto d_HT_634ccc640bf0 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_634ccc66fcf0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_634ccc688bb0, d_HT_634ccc640bf0.ref(cuco::insert), d_HT_634ccc688bb0.ref(cuco::for_each), d_lineorder__lo_discount, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
size_t COUNT634ccc640bf0 = d_HT_634ccc640bf0.size();
thrust::device_vector<int64_t> keys_634ccc640bf0(COUNT634ccc640bf0), vals_634ccc640bf0(COUNT634ccc640bf0);
d_HT_634ccc640bf0.retrieve_all(keys_634ccc640bf0.begin(), vals_634ccc640bf0.begin());
d_HT_634ccc640bf0.clear();
int64_t* raw_keys634ccc640bf0 = thrust::raw_pointer_cast(keys_634ccc640bf0.data());
insertKeys<<<std::ceil((float)COUNT634ccc640bf0/128.), 128>>>(raw_keys634ccc640bf0, d_HT_634ccc640bf0.ref(cuco::insert), COUNT634ccc640bf0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT634ccc640bf0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT634ccc640bf0);
main_634ccc66fcf0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_634ccc688bb0, d_HT_634ccc640bf0.ref(cuco::find), d_HT_634ccc688bb0.ref(cuco::for_each), d_aggr0__tmp_attr0, d_lineorder__lo_discount, d_lineorder__lo_extendedprice, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
//Materialize count
uint64_t* d_COUNT634ccc652f60;
hipMalloc(&d_COUNT634ccc652f60, sizeof(uint64_t));
hipMemset(d_COUNT634ccc652f60, 0, sizeof(uint64_t));
count_634ccc698fe0<<<std::ceil((float)COUNT634ccc640bf0/128.), 128>>>(COUNT634ccc640bf0, d_COUNT634ccc652f60);
uint64_t COUNT634ccc652f60;
hipMemcpy(&COUNT634ccc652f60, d_COUNT634ccc652f60, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX634ccc652f60;
hipMalloc(&d_MAT_IDX634ccc652f60, sizeof(uint64_t));
hipMemset(d_MAT_IDX634ccc652f60, 0, sizeof(uint64_t));
auto MAT634ccc652f60aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT634ccc652f60);
DBDecimalType* d_MAT634ccc652f60aggr0__tmp_attr0;
hipMalloc(&d_MAT634ccc652f60aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT634ccc652f60);
main_634ccc698fe0<<<std::ceil((float)COUNT634ccc640bf0/128.), 128>>>(COUNT634ccc640bf0, d_MAT634ccc652f60aggr0__tmp_attr0, d_MAT_IDX634ccc652f60, d_aggr0__tmp_attr0);
hipMemcpy(MAT634ccc652f60aggr0__tmp_attr0, d_MAT634ccc652f60aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT634ccc652f60, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT634ccc652f60; i++) { std::cout << "" << MAT634ccc652f60aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_634ccc688bb0);
hipFree(d_BUF_IDX_634ccc688bb0);
hipFree(d_COUNT634ccc688bb0);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT634ccc652f60);
hipFree(d_MAT634ccc652f60aggr0__tmp_attr0);
hipFree(d_MAT_IDX634ccc652f60);
free(MAT634ccc652f60aggr0__tmp_attr0);
}