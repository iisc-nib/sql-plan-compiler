#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5bf4ac2e2c80(uint64_t* COUNT5bf4ac2d6980, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5bf4ac2d6980, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5bf4ac2e2c80(uint64_t* BUF_5bf4ac2d6980, uint64_t* BUF_IDX_5bf4ac2d6980, HASHTABLE_INSERT HT_5bf4ac2d6980, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5bf4ac2d6980 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5bf4ac2d6980 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5bf4ac2d6980 = atomicAdd((int*)BUF_IDX_5bf4ac2d6980, 1);
HT_5bf4ac2d6980.insert(cuco::pair{KEY_5bf4ac2d6980, buf_idx_5bf4ac2d6980});
BUF_5bf4ac2d6980[buf_idx_5bf4ac2d6980 * 1 + 0] = tid;
}
__global__ void count_5bf4ac27c9a0(uint64_t* COUNT5bf4ac2d5a20, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5bf4ac2d5a20, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5bf4ac27c9a0(uint64_t* BUF_5bf4ac2d5a20, uint64_t* BUF_IDX_5bf4ac2d5a20, HASHTABLE_INSERT HT_5bf4ac2d5a20, DBI32Type* customer__c_custkey, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5bf4ac2d5a20 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5bf4ac2d5a20 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_5bf4ac2d5a20 = atomicAdd((int*)BUF_IDX_5bf4ac2d5a20, 1);
HT_5bf4ac2d5a20.insert(cuco::pair{KEY_5bf4ac2d5a20, buf_idx_5bf4ac2d5a20});
BUF_5bf4ac2d5a20[buf_idx_5bf4ac2d5a20 * 1 + 0] = tid;
}
__global__ void count_5bf4ac2e4f90(uint64_t* COUNT5bf4ac2d73e0, DBStringType* part__p_mfgr, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_mfgr = part__p_mfgr[tid];
if (!((evaluatePredicate(reg_part__p_mfgr, "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr, "MFGR#2", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5bf4ac2d73e0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5bf4ac2e4f90(uint64_t* BUF_5bf4ac2d73e0, uint64_t* BUF_IDX_5bf4ac2d73e0, HASHTABLE_INSERT HT_5bf4ac2d73e0, DBStringType* part__p_mfgr, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_mfgr = part__p_mfgr[tid];
if (!((evaluatePredicate(reg_part__p_mfgr, "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr, "MFGR#2", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5bf4ac2d73e0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_5bf4ac2d73e0 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_5bf4ac2d73e0 = atomicAdd((int*)BUF_IDX_5bf4ac2d73e0, 1);
HT_5bf4ac2d73e0.insert(cuco::pair{KEY_5bf4ac2d73e0, buf_idx_5bf4ac2d73e0});
BUF_5bf4ac2d73e0[buf_idx_5bf4ac2d73e0 * 1 + 0] = tid;
}
__global__ void count_5bf4ac27cbe0(uint64_t* COUNT5bf4ac2b6330, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5bf4ac2b6330, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5bf4ac27cbe0(uint64_t* BUF_5bf4ac2b6330, uint64_t* BUF_IDX_5bf4ac2b6330, HASHTABLE_INSERT HT_5bf4ac2b6330, DBI32Type* date__d_datekey, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5bf4ac2b6330 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5bf4ac2b6330 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_5bf4ac2b6330 = atomicAdd((int*)BUF_IDX_5bf4ac2b6330, 1);
HT_5bf4ac2b6330.insert(cuco::pair{KEY_5bf4ac2b6330, buf_idx_5bf4ac2b6330});
BUF_5bf4ac2b6330[buf_idx_5bf4ac2b6330 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5bf4ac2e71d0(uint64_t* BUF_5bf4ac2b6330, uint64_t* BUF_5bf4ac2d5a20, uint64_t* BUF_5bf4ac2d6980, uint64_t* BUF_5bf4ac2d73e0, HASHTABLE_INSERT HT_5bf4ac28be20, HASHTABLE_PROBE HT_5bf4ac2b6330, HASHTABLE_PROBE HT_5bf4ac2d5a20, HASHTABLE_PROBE HT_5bf4ac2d6980, HASHTABLE_PROBE HT_5bf4ac2d73e0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5bf4ac2d6980 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5bf4ac2d6980 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5bf4ac2d6980.for_each(KEY_5bf4ac2d6980, [&] __device__ (auto const SLOT_5bf4ac2d6980) {

auto const [slot_first5bf4ac2d6980, slot_second5bf4ac2d6980] = SLOT_5bf4ac2d6980;
if (!(true)) return;
uint64_t KEY_5bf4ac2d5a20 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5bf4ac2d5a20 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_5bf4ac2d5a20.for_each(KEY_5bf4ac2d5a20, [&] __device__ (auto const SLOT_5bf4ac2d5a20) {

auto const [slot_first5bf4ac2d5a20, slot_second5bf4ac2d5a20] = SLOT_5bf4ac2d5a20;
if (!(true)) return;
uint64_t KEY_5bf4ac2d73e0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5bf4ac2d73e0 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5bf4ac2d73e0.for_each(KEY_5bf4ac2d73e0, [&] __device__ (auto const SLOT_5bf4ac2d73e0) {

auto const [slot_first5bf4ac2d73e0, slot_second5bf4ac2d73e0] = SLOT_5bf4ac2d73e0;
if (!(true)) return;
uint64_t KEY_5bf4ac2b6330 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5bf4ac2b6330 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5bf4ac2b6330.for_each(KEY_5bf4ac2b6330, [&] __device__ (auto const SLOT_5bf4ac2b6330) {

auto const [slot_first5bf4ac2b6330, slot_second5bf4ac2b6330] = SLOT_5bf4ac2b6330;
if (!(true)) return;
uint64_t KEY_5bf4ac28be20 = 0;
auto reg_date__d_year = date__d_year[BUF_5bf4ac2b6330[slot_second5bf4ac2b6330 * 1 + 0]];

KEY_5bf4ac28be20 |= reg_date__d_year;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[BUF_5bf4ac2d5a20[slot_second5bf4ac2d5a20 * 1 + 0]];
KEY_5bf4ac28be20 <<= 16;
KEY_5bf4ac28be20 |= reg_customer__c_nation_encoded;
//Create aggregation hash table
HT_5bf4ac28be20.insert(cuco::pair{KEY_5bf4ac28be20, 1});
});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5bf4ac2e71d0(uint64_t* BUF_5bf4ac2b6330, uint64_t* BUF_5bf4ac2d5a20, uint64_t* BUF_5bf4ac2d6980, uint64_t* BUF_5bf4ac2d73e0, HASHTABLE_FIND HT_5bf4ac28be20, HASHTABLE_PROBE HT_5bf4ac2b6330, HASHTABLE_PROBE HT_5bf4ac2d5a20, HASHTABLE_PROBE HT_5bf4ac2d6980, HASHTABLE_PROBE HT_5bf4ac2d73e0, DBI16Type* KEY_5bf4ac28be20customer__c_nation_encoded, DBI32Type* KEY_5bf4ac28be20date__d_year, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, DBDecimalType* lineorder__lo_supplycost, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5bf4ac2d6980 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5bf4ac2d6980 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5bf4ac2d6980.for_each(KEY_5bf4ac2d6980, [&] __device__ (auto const SLOT_5bf4ac2d6980) {
auto const [slot_first5bf4ac2d6980, slot_second5bf4ac2d6980] = SLOT_5bf4ac2d6980;
if (!(true)) return;
uint64_t KEY_5bf4ac2d5a20 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5bf4ac2d5a20 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_5bf4ac2d5a20.for_each(KEY_5bf4ac2d5a20, [&] __device__ (auto const SLOT_5bf4ac2d5a20) {
auto const [slot_first5bf4ac2d5a20, slot_second5bf4ac2d5a20] = SLOT_5bf4ac2d5a20;
if (!(true)) return;
uint64_t KEY_5bf4ac2d73e0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5bf4ac2d73e0 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5bf4ac2d73e0.for_each(KEY_5bf4ac2d73e0, [&] __device__ (auto const SLOT_5bf4ac2d73e0) {
auto const [slot_first5bf4ac2d73e0, slot_second5bf4ac2d73e0] = SLOT_5bf4ac2d73e0;
if (!(true)) return;
uint64_t KEY_5bf4ac2b6330 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5bf4ac2b6330 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5bf4ac2b6330.for_each(KEY_5bf4ac2b6330, [&] __device__ (auto const SLOT_5bf4ac2b6330) {
auto const [slot_first5bf4ac2b6330, slot_second5bf4ac2b6330] = SLOT_5bf4ac2b6330;
if (!(true)) return;
uint64_t KEY_5bf4ac28be20 = 0;
auto reg_date__d_year = date__d_year[BUF_5bf4ac2b6330[slot_second5bf4ac2b6330 * 1 + 0]];

KEY_5bf4ac28be20 |= reg_date__d_year;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[BUF_5bf4ac2d5a20[slot_second5bf4ac2d5a20 * 1 + 0]];
KEY_5bf4ac28be20 <<= 16;
KEY_5bf4ac28be20 |= reg_customer__c_nation_encoded;
//Aggregate in hashtable
auto buf_idx_5bf4ac28be20 = HT_5bf4ac28be20.find(KEY_5bf4ac28be20)->second;
auto reg_lineorder__lo_supplycost = lineorder__lo_supplycost[tid];
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_revenue) - (reg_lineorder__lo_supplycost);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5bf4ac28be20], reg_map0__tmp_attr1);
KEY_5bf4ac28be20date__d_year[buf_idx_5bf4ac28be20] = reg_date__d_year;
KEY_5bf4ac28be20customer__c_nation_encoded[buf_idx_5bf4ac28be20] = reg_customer__c_nation_encoded;
});
});
});
});
}
__global__ void count_5bf4ac2fa5f0(uint64_t* COUNT5bf4ac26b160, size_t COUNT5bf4ac28be20) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5bf4ac28be20) return;
//Materialize count
atomicAdd((int*)COUNT5bf4ac26b160, 1);
}
__global__ void main_5bf4ac2fa5f0(size_t COUNT5bf4ac28be20, DBDecimalType* MAT5bf4ac26b160aggr0__tmp_attr0, DBI16Type* MAT5bf4ac26b160customer__c_nation_encoded, DBI32Type* MAT5bf4ac26b160date__d_year, uint64_t* MAT_IDX5bf4ac26b160, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5bf4ac28be20) return;
//Materialize buffers
auto mat_idx5bf4ac26b160 = atomicAdd((int*)MAT_IDX5bf4ac26b160, 1);
auto reg_date__d_year = date__d_year[tid];
MAT5bf4ac26b160date__d_year[mat_idx5bf4ac26b160] = reg_date__d_year;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[tid];
MAT5bf4ac26b160customer__c_nation_encoded[mat_idx5bf4ac26b160] = reg_customer__c_nation_encoded;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5bf4ac26b160aggr0__tmp_attr0[mat_idx5bf4ac26b160] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map) {
//Materialize count
uint64_t* d_COUNT5bf4ac2d6980;
hipMalloc(&d_COUNT5bf4ac2d6980, sizeof(uint64_t));
hipMemset(d_COUNT5bf4ac2d6980, 0, sizeof(uint64_t));
count_5bf4ac2e2c80<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT5bf4ac2d6980, d_supplier__s_region, supplier_size);
uint64_t COUNT5bf4ac2d6980;
hipMemcpy(&COUNT5bf4ac2d6980, d_COUNT5bf4ac2d6980, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5bf4ac2d6980;
hipMalloc(&d_BUF_IDX_5bf4ac2d6980, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5bf4ac2d6980, 0, sizeof(uint64_t));
uint64_t* d_BUF_5bf4ac2d6980;
hipMalloc(&d_BUF_5bf4ac2d6980, sizeof(uint64_t) * COUNT5bf4ac2d6980 * 1);
auto d_HT_5bf4ac2d6980 = cuco::experimental::static_multimap{ (int)COUNT5bf4ac2d6980*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5bf4ac2e2c80<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_5bf4ac2d6980, d_BUF_IDX_5bf4ac2d6980, d_HT_5bf4ac2d6980.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5bf4ac2d5a20;
hipMalloc(&d_COUNT5bf4ac2d5a20, sizeof(uint64_t));
hipMemset(d_COUNT5bf4ac2d5a20, 0, sizeof(uint64_t));
count_5bf4ac27c9a0<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT5bf4ac2d5a20, d_customer__c_region, customer_size);
uint64_t COUNT5bf4ac2d5a20;
hipMemcpy(&COUNT5bf4ac2d5a20, d_COUNT5bf4ac2d5a20, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5bf4ac2d5a20;
hipMalloc(&d_BUF_IDX_5bf4ac2d5a20, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5bf4ac2d5a20, 0, sizeof(uint64_t));
uint64_t* d_BUF_5bf4ac2d5a20;
hipMalloc(&d_BUF_5bf4ac2d5a20, sizeof(uint64_t) * COUNT5bf4ac2d5a20 * 1);
auto d_HT_5bf4ac2d5a20 = cuco::experimental::static_multimap{ (int)COUNT5bf4ac2d5a20*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5bf4ac27c9a0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5bf4ac2d5a20, d_BUF_IDX_5bf4ac2d5a20, d_HT_5bf4ac2d5a20.ref(cuco::insert), d_customer__c_custkey, d_customer__c_region, customer_size);
//Materialize count
uint64_t* d_COUNT5bf4ac2d73e0;
hipMalloc(&d_COUNT5bf4ac2d73e0, sizeof(uint64_t));
hipMemset(d_COUNT5bf4ac2d73e0, 0, sizeof(uint64_t));
count_5bf4ac2e4f90<<<std::ceil((float)part_size/32.), 32>>>(d_COUNT5bf4ac2d73e0, d_part__p_mfgr, part_size);
uint64_t COUNT5bf4ac2d73e0;
hipMemcpy(&COUNT5bf4ac2d73e0, d_COUNT5bf4ac2d73e0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5bf4ac2d73e0;
hipMalloc(&d_BUF_IDX_5bf4ac2d73e0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5bf4ac2d73e0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5bf4ac2d73e0;
hipMalloc(&d_BUF_5bf4ac2d73e0, sizeof(uint64_t) * COUNT5bf4ac2d73e0 * 1);
auto d_HT_5bf4ac2d73e0 = cuco::experimental::static_multimap{ (int)COUNT5bf4ac2d73e0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5bf4ac2e4f90<<<std::ceil((float)part_size/32.), 32>>>(d_BUF_5bf4ac2d73e0, d_BUF_IDX_5bf4ac2d73e0, d_HT_5bf4ac2d73e0.ref(cuco::insert), d_part__p_mfgr, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT5bf4ac2b6330;
hipMalloc(&d_COUNT5bf4ac2b6330, sizeof(uint64_t));
hipMemset(d_COUNT5bf4ac2b6330, 0, sizeof(uint64_t));
count_5bf4ac27cbe0<<<std::ceil((float)date_size/32.), 32>>>(d_COUNT5bf4ac2b6330, date_size);
uint64_t COUNT5bf4ac2b6330;
hipMemcpy(&COUNT5bf4ac2b6330, d_COUNT5bf4ac2b6330, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5bf4ac2b6330;
hipMalloc(&d_BUF_IDX_5bf4ac2b6330, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5bf4ac2b6330, 0, sizeof(uint64_t));
uint64_t* d_BUF_5bf4ac2b6330;
hipMalloc(&d_BUF_5bf4ac2b6330, sizeof(uint64_t) * COUNT5bf4ac2b6330 * 1);
auto d_HT_5bf4ac2b6330 = cuco::experimental::static_multimap{ (int)COUNT5bf4ac2b6330*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5bf4ac27cbe0<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_5bf4ac2b6330, d_BUF_IDX_5bf4ac2b6330, d_HT_5bf4ac2b6330.ref(cuco::insert), d_date__d_datekey, date_size);
//Create aggregation hash table
auto d_HT_5bf4ac28be20 = cuco::static_map{ (int)87950*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5bf4ac2e71d0<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_5bf4ac2b6330, d_BUF_5bf4ac2d5a20, d_BUF_5bf4ac2d6980, d_BUF_5bf4ac2d73e0, d_HT_5bf4ac28be20.ref(cuco::insert), d_HT_5bf4ac2b6330.ref(cuco::for_each), d_HT_5bf4ac2d5a20.ref(cuco::for_each), d_HT_5bf4ac2d6980.ref(cuco::for_each), d_HT_5bf4ac2d73e0.ref(cuco::for_each), d_customer__c_nation_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
size_t COUNT5bf4ac28be20 = d_HT_5bf4ac28be20.size();
std::cout << "COUNT5bf4ac28be20: " << COUNT5bf4ac28be20 << std::endl;
thrust::device_vector<int64_t> keys_5bf4ac28be20(COUNT5bf4ac28be20), vals_5bf4ac28be20(COUNT5bf4ac28be20);
d_HT_5bf4ac28be20.retrieve_all(keys_5bf4ac28be20.begin(), vals_5bf4ac28be20.begin());
d_HT_5bf4ac28be20.clear();
int64_t* raw_keys5bf4ac28be20 = thrust::raw_pointer_cast(keys_5bf4ac28be20.data());
insertKeys<<<std::ceil((float)COUNT5bf4ac28be20/32.), 32>>>(raw_keys5bf4ac28be20, d_HT_5bf4ac28be20.ref(cuco::insert), COUNT5bf4ac28be20);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5bf4ac28be20);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5bf4ac28be20);
DBI32Type* d_KEY_5bf4ac28be20date__d_year;
hipMalloc(&d_KEY_5bf4ac28be20date__d_year, sizeof(DBI32Type) * COUNT5bf4ac28be20);
hipMemset(d_KEY_5bf4ac28be20date__d_year, 0, sizeof(DBI32Type) * COUNT5bf4ac28be20);
DBI16Type* d_KEY_5bf4ac28be20customer__c_nation_encoded;
hipMalloc(&d_KEY_5bf4ac28be20customer__c_nation_encoded, sizeof(DBI16Type) * COUNT5bf4ac28be20);
hipMemset(d_KEY_5bf4ac28be20customer__c_nation_encoded, 0, sizeof(DBI16Type) * COUNT5bf4ac28be20);
main_5bf4ac2e71d0<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_5bf4ac2b6330, d_BUF_5bf4ac2d5a20, d_BUF_5bf4ac2d6980, d_BUF_5bf4ac2d73e0, d_HT_5bf4ac28be20.ref(cuco::find), d_HT_5bf4ac2b6330.ref(cuco::for_each), d_HT_5bf4ac2d5a20.ref(cuco::for_each), d_HT_5bf4ac2d6980.ref(cuco::for_each), d_HT_5bf4ac2d73e0.ref(cuco::for_each), d_KEY_5bf4ac28be20customer__c_nation_encoded, d_KEY_5bf4ac28be20date__d_year, d_aggr0__tmp_attr0, d_customer__c_nation_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, d_lineorder__lo_supplycost, lineorder_size);
//Materialize count
uint64_t* d_COUNT5bf4ac26b160;
hipMalloc(&d_COUNT5bf4ac26b160, sizeof(uint64_t));
hipMemset(d_COUNT5bf4ac26b160, 0, sizeof(uint64_t));
count_5bf4ac2fa5f0<<<std::ceil((float)COUNT5bf4ac28be20/32.), 32>>>(d_COUNT5bf4ac26b160, COUNT5bf4ac28be20);
uint64_t COUNT5bf4ac26b160;
hipMemcpy(&COUNT5bf4ac26b160, d_COUNT5bf4ac26b160, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5bf4ac26b160;
hipMalloc(&d_MAT_IDX5bf4ac26b160, sizeof(uint64_t));
hipMemset(d_MAT_IDX5bf4ac26b160, 0, sizeof(uint64_t));
auto MAT5bf4ac26b160date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5bf4ac26b160);
DBI32Type* d_MAT5bf4ac26b160date__d_year;
hipMalloc(&d_MAT5bf4ac26b160date__d_year, sizeof(DBI32Type) * COUNT5bf4ac26b160);
auto MAT5bf4ac26b160customer__c_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5bf4ac26b160);
DBI16Type* d_MAT5bf4ac26b160customer__c_nation_encoded;
hipMalloc(&d_MAT5bf4ac26b160customer__c_nation_encoded, sizeof(DBI16Type) * COUNT5bf4ac26b160);
auto MAT5bf4ac26b160aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5bf4ac26b160);
DBDecimalType* d_MAT5bf4ac26b160aggr0__tmp_attr0;
hipMalloc(&d_MAT5bf4ac26b160aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5bf4ac26b160);
main_5bf4ac2fa5f0<<<std::ceil((float)COUNT5bf4ac28be20/32.), 32>>>(COUNT5bf4ac28be20, d_MAT5bf4ac26b160aggr0__tmp_attr0, d_MAT5bf4ac26b160customer__c_nation_encoded, d_MAT5bf4ac26b160date__d_year, d_MAT_IDX5bf4ac26b160, d_aggr0__tmp_attr0, d_KEY_5bf4ac28be20customer__c_nation_encoded, d_KEY_5bf4ac28be20date__d_year);
hipMemcpy(MAT5bf4ac26b160date__d_year, d_MAT5bf4ac26b160date__d_year, sizeof(DBI32Type) * COUNT5bf4ac26b160, hipMemcpyDeviceToHost);
hipMemcpy(MAT5bf4ac26b160customer__c_nation_encoded, d_MAT5bf4ac26b160customer__c_nation_encoded, sizeof(DBI16Type) * COUNT5bf4ac26b160, hipMemcpyDeviceToHost);
hipMemcpy(MAT5bf4ac26b160aggr0__tmp_attr0, d_MAT5bf4ac26b160aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5bf4ac26b160, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5bf4ac26b160; i++) { std::cout << MAT5bf4ac26b160date__d_year[i] << "\t";
std::cout << customer__c_nation_map[MAT5bf4ac26b160customer__c_nation_encoded[i]] << "\t";
std::cout << MAT5bf4ac26b160aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_5bf4ac2d6980);
hipFree(d_BUF_IDX_5bf4ac2d6980);
hipFree(d_COUNT5bf4ac2d6980);
hipFree(d_BUF_5bf4ac2d5a20);
hipFree(d_BUF_IDX_5bf4ac2d5a20);
hipFree(d_COUNT5bf4ac2d5a20);
hipFree(d_BUF_5bf4ac2d73e0);
hipFree(d_BUF_IDX_5bf4ac2d73e0);
hipFree(d_COUNT5bf4ac2d73e0);
hipFree(d_BUF_5bf4ac2b6330);
hipFree(d_BUF_IDX_5bf4ac2b6330);
hipFree(d_COUNT5bf4ac2b6330);
hipFree(d_KEY_5bf4ac28be20customer__c_nation_encoded);
hipFree(d_KEY_5bf4ac28be20date__d_year);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5bf4ac26b160);
hipFree(d_MAT5bf4ac26b160aggr0__tmp_attr0);
hipFree(d_MAT5bf4ac26b160customer__c_nation_encoded);
hipFree(d_MAT5bf4ac26b160date__d_year);
hipFree(d_MAT_IDX5bf4ac26b160);
free(MAT5bf4ac26b160aggr0__tmp_attr0);
free(MAT5bf4ac26b160customer__c_nation_encoded);
free(MAT5bf4ac26b160date__d_year);
}