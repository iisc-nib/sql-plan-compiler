#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_633d2e1a4e10(uint64_t* COUNT633d2e196490, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT633d2e196490, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_633d2e1a4e10(uint64_t* BUF_633d2e196490, uint64_t* BUF_IDX_633d2e196490, HASHTABLE_INSERT HT_633d2e196490, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_633d2e196490 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_633d2e196490 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_633d2e196490 = atomicAdd((int*)BUF_IDX_633d2e196490, 1);
HT_633d2e196490.insert(cuco::pair{KEY_633d2e196490, buf_idx_633d2e196490});
BUF_633d2e196490[buf_idx_633d2e196490 * 1 + 0] = tid;
}
__global__ void count_633d2e13f450(uint64_t* COUNT633d2e178960, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT633d2e178960, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_633d2e13f450(uint64_t* BUF_633d2e178960, uint64_t* BUF_IDX_633d2e178960, HASHTABLE_INSERT HT_633d2e178960, DBI32Type* customer__c_custkey, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_633d2e178960 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_633d2e178960 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_633d2e178960 = atomicAdd((int*)BUF_IDX_633d2e178960, 1);
HT_633d2e178960.insert(cuco::pair{KEY_633d2e178960, buf_idx_633d2e178960});
BUF_633d2e178960[buf_idx_633d2e178960 * 1 + 0] = tid;
}
__global__ void count_633d2e1a7630(uint64_t* COUNT633d2e19b8a0, DBStringType* part__p_mfgr, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_mfgr = part__p_mfgr[tid];
if (!((evaluatePredicate(reg_part__p_mfgr, "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr, "MFGR#2", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT633d2e19b8a0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_633d2e1a7630(uint64_t* BUF_633d2e19b8a0, uint64_t* BUF_IDX_633d2e19b8a0, HASHTABLE_INSERT HT_633d2e19b8a0, DBStringType* part__p_mfgr, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_mfgr = part__p_mfgr[tid];
if (!((evaluatePredicate(reg_part__p_mfgr, "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr, "MFGR#2", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_633d2e19b8a0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_633d2e19b8a0 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_633d2e19b8a0 = atomicAdd((int*)BUF_IDX_633d2e19b8a0, 1);
HT_633d2e19b8a0.insert(cuco::pair{KEY_633d2e19b8a0, buf_idx_633d2e19b8a0});
BUF_633d2e19b8a0[buf_idx_633d2e19b8a0 * 1 + 0] = tid;
}
__global__ void count_633d2e13f690(uint64_t* COUNT633d2e19b0b0, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT633d2e19b0b0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_633d2e13f690(uint64_t* BUF_633d2e19b0b0, uint64_t* BUF_IDX_633d2e19b0b0, HASHTABLE_INSERT HT_633d2e19b0b0, DBI32Type* date__d_datekey, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_633d2e19b0b0 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_633d2e19b0b0 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_633d2e19b0b0 = atomicAdd((int*)BUF_IDX_633d2e19b0b0, 1);
HT_633d2e19b0b0.insert(cuco::pair{KEY_633d2e19b0b0, buf_idx_633d2e19b0b0});
BUF_633d2e19b0b0[buf_idx_633d2e19b0b0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_633d2e1a9e90(uint64_t* BUF_633d2e178960, uint64_t* BUF_633d2e196490, uint64_t* BUF_633d2e19b0b0, uint64_t* BUF_633d2e19b8a0, HASHTABLE_INSERT HT_633d2e14e9b0, HASHTABLE_PROBE HT_633d2e178960, HASHTABLE_PROBE HT_633d2e196490, HASHTABLE_PROBE HT_633d2e19b0b0, HASHTABLE_PROBE HT_633d2e19b8a0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_633d2e196490 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_633d2e196490 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_633d2e196490.for_each(KEY_633d2e196490, [&] __device__ (auto const SLOT_633d2e196490) {

auto const [slot_first633d2e196490, slot_second633d2e196490] = SLOT_633d2e196490;
if (!(true)) return;
uint64_t KEY_633d2e178960 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_633d2e178960 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_633d2e178960.for_each(KEY_633d2e178960, [&] __device__ (auto const SLOT_633d2e178960) {

auto const [slot_first633d2e178960, slot_second633d2e178960] = SLOT_633d2e178960;
if (!(true)) return;
uint64_t KEY_633d2e19b8a0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_633d2e19b8a0 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_633d2e19b8a0.for_each(KEY_633d2e19b8a0, [&] __device__ (auto const SLOT_633d2e19b8a0) {

auto const [slot_first633d2e19b8a0, slot_second633d2e19b8a0] = SLOT_633d2e19b8a0;
if (!(true)) return;
uint64_t KEY_633d2e19b0b0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_633d2e19b0b0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_633d2e19b0b0.for_each(KEY_633d2e19b0b0, [&] __device__ (auto const SLOT_633d2e19b0b0) {

auto const [slot_first633d2e19b0b0, slot_second633d2e19b0b0] = SLOT_633d2e19b0b0;
if (!(true)) return;
uint64_t KEY_633d2e14e9b0 = 0;
auto reg_date__d_year = date__d_year[BUF_633d2e19b0b0[slot_second633d2e19b0b0 * 1 + 0]];

KEY_633d2e14e9b0 |= reg_date__d_year;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[BUF_633d2e178960[slot_second633d2e178960 * 1 + 0]];
KEY_633d2e14e9b0 <<= 16;
KEY_633d2e14e9b0 |= reg_customer__c_nation_encoded;
//Create aggregation hash table
HT_633d2e14e9b0.insert(cuco::pair{KEY_633d2e14e9b0, 1});
});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_633d2e1a9e90(uint64_t* BUF_633d2e178960, uint64_t* BUF_633d2e196490, uint64_t* BUF_633d2e19b0b0, uint64_t* BUF_633d2e19b8a0, HASHTABLE_FIND HT_633d2e14e9b0, HASHTABLE_PROBE HT_633d2e178960, HASHTABLE_PROBE HT_633d2e196490, HASHTABLE_PROBE HT_633d2e19b0b0, HASHTABLE_PROBE HT_633d2e19b8a0, DBI16Type* KEY_633d2e14e9b0customer__c_nation_encoded, DBI32Type* KEY_633d2e14e9b0date__d_year, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, DBDecimalType* lineorder__lo_supplycost, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_633d2e196490 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_633d2e196490 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_633d2e196490.for_each(KEY_633d2e196490, [&] __device__ (auto const SLOT_633d2e196490) {
auto const [slot_first633d2e196490, slot_second633d2e196490] = SLOT_633d2e196490;
if (!(true)) return;
uint64_t KEY_633d2e178960 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_633d2e178960 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_633d2e178960.for_each(KEY_633d2e178960, [&] __device__ (auto const SLOT_633d2e178960) {
auto const [slot_first633d2e178960, slot_second633d2e178960] = SLOT_633d2e178960;
if (!(true)) return;
uint64_t KEY_633d2e19b8a0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_633d2e19b8a0 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_633d2e19b8a0.for_each(KEY_633d2e19b8a0, [&] __device__ (auto const SLOT_633d2e19b8a0) {
auto const [slot_first633d2e19b8a0, slot_second633d2e19b8a0] = SLOT_633d2e19b8a0;
if (!(true)) return;
uint64_t KEY_633d2e19b0b0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_633d2e19b0b0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_633d2e19b0b0.for_each(KEY_633d2e19b0b0, [&] __device__ (auto const SLOT_633d2e19b0b0) {
auto const [slot_first633d2e19b0b0, slot_second633d2e19b0b0] = SLOT_633d2e19b0b0;
if (!(true)) return;
uint64_t KEY_633d2e14e9b0 = 0;
auto reg_date__d_year = date__d_year[BUF_633d2e19b0b0[slot_second633d2e19b0b0 * 1 + 0]];

KEY_633d2e14e9b0 |= reg_date__d_year;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[BUF_633d2e178960[slot_second633d2e178960 * 1 + 0]];
KEY_633d2e14e9b0 <<= 16;
KEY_633d2e14e9b0 |= reg_customer__c_nation_encoded;
//Aggregate in hashtable
auto buf_idx_633d2e14e9b0 = HT_633d2e14e9b0.find(KEY_633d2e14e9b0)->second;
auto reg_lineorder__lo_supplycost = lineorder__lo_supplycost[tid];
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_revenue) - (reg_lineorder__lo_supplycost);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_633d2e14e9b0], reg_map0__tmp_attr1);
KEY_633d2e14e9b0date__d_year[buf_idx_633d2e14e9b0] = reg_date__d_year;
KEY_633d2e14e9b0customer__c_nation_encoded[buf_idx_633d2e14e9b0] = reg_customer__c_nation_encoded;
});
});
});
});
}
__global__ void count_633d2e1bd580(uint64_t* COUNT633d2e12db50, size_t COUNT633d2e14e9b0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT633d2e14e9b0) return;
//Materialize count
atomicAdd((int*)COUNT633d2e12db50, 1);
}
__global__ void main_633d2e1bd580(size_t COUNT633d2e14e9b0, DBDecimalType* MAT633d2e12db50aggr0__tmp_attr0, DBI16Type* MAT633d2e12db50customer__c_nation_encoded, DBI32Type* MAT633d2e12db50date__d_year, uint64_t* MAT_IDX633d2e12db50, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT633d2e14e9b0) return;
//Materialize buffers
auto mat_idx633d2e12db50 = atomicAdd((int*)MAT_IDX633d2e12db50, 1);
auto reg_date__d_year = date__d_year[tid];
MAT633d2e12db50date__d_year[mat_idx633d2e12db50] = reg_date__d_year;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[tid];
MAT633d2e12db50customer__c_nation_encoded[mat_idx633d2e12db50] = reg_customer__c_nation_encoded;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT633d2e12db50aggr0__tmp_attr0[mat_idx633d2e12db50] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT633d2e196490;
hipMalloc(&d_COUNT633d2e196490, sizeof(uint64_t));
hipMemset(d_COUNT633d2e196490, 0, sizeof(uint64_t));
count_633d2e1a4e10<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT633d2e196490, d_supplier__s_region, supplier_size);
uint64_t COUNT633d2e196490;
hipMemcpy(&COUNT633d2e196490, d_COUNT633d2e196490, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_633d2e196490;
hipMalloc(&d_BUF_IDX_633d2e196490, sizeof(uint64_t));
hipMemset(d_BUF_IDX_633d2e196490, 0, sizeof(uint64_t));
uint64_t* d_BUF_633d2e196490;
hipMalloc(&d_BUF_633d2e196490, sizeof(uint64_t) * COUNT633d2e196490 * 1);
auto d_HT_633d2e196490 = cuco::experimental::static_multimap{ (int)COUNT633d2e196490*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_633d2e1a4e10<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_633d2e196490, d_BUF_IDX_633d2e196490, d_HT_633d2e196490.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT633d2e178960;
hipMalloc(&d_COUNT633d2e178960, sizeof(uint64_t));
hipMemset(d_COUNT633d2e178960, 0, sizeof(uint64_t));
count_633d2e13f450<<<std::ceil((float)customer_size/128.), 128>>>(d_COUNT633d2e178960, d_customer__c_region, customer_size);
uint64_t COUNT633d2e178960;
hipMemcpy(&COUNT633d2e178960, d_COUNT633d2e178960, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_633d2e178960;
hipMalloc(&d_BUF_IDX_633d2e178960, sizeof(uint64_t));
hipMemset(d_BUF_IDX_633d2e178960, 0, sizeof(uint64_t));
uint64_t* d_BUF_633d2e178960;
hipMalloc(&d_BUF_633d2e178960, sizeof(uint64_t) * COUNT633d2e178960 * 1);
auto d_HT_633d2e178960 = cuco::experimental::static_multimap{ (int)COUNT633d2e178960*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_633d2e13f450<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_633d2e178960, d_BUF_IDX_633d2e178960, d_HT_633d2e178960.ref(cuco::insert), d_customer__c_custkey, d_customer__c_region, customer_size);
//Materialize count
uint64_t* d_COUNT633d2e19b8a0;
hipMalloc(&d_COUNT633d2e19b8a0, sizeof(uint64_t));
hipMemset(d_COUNT633d2e19b8a0, 0, sizeof(uint64_t));
count_633d2e1a7630<<<std::ceil((float)part_size/128.), 128>>>(d_COUNT633d2e19b8a0, d_part__p_mfgr, part_size);
uint64_t COUNT633d2e19b8a0;
hipMemcpy(&COUNT633d2e19b8a0, d_COUNT633d2e19b8a0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_633d2e19b8a0;
hipMalloc(&d_BUF_IDX_633d2e19b8a0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_633d2e19b8a0, 0, sizeof(uint64_t));
uint64_t* d_BUF_633d2e19b8a0;
hipMalloc(&d_BUF_633d2e19b8a0, sizeof(uint64_t) * COUNT633d2e19b8a0 * 1);
auto d_HT_633d2e19b8a0 = cuco::experimental::static_multimap{ (int)COUNT633d2e19b8a0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_633d2e1a7630<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_633d2e19b8a0, d_BUF_IDX_633d2e19b8a0, d_HT_633d2e19b8a0.ref(cuco::insert), d_part__p_mfgr, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT633d2e19b0b0;
hipMalloc(&d_COUNT633d2e19b0b0, sizeof(uint64_t));
hipMemset(d_COUNT633d2e19b0b0, 0, sizeof(uint64_t));
count_633d2e13f690<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT633d2e19b0b0, date_size);
uint64_t COUNT633d2e19b0b0;
hipMemcpy(&COUNT633d2e19b0b0, d_COUNT633d2e19b0b0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_633d2e19b0b0;
hipMalloc(&d_BUF_IDX_633d2e19b0b0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_633d2e19b0b0, 0, sizeof(uint64_t));
uint64_t* d_BUF_633d2e19b0b0;
hipMalloc(&d_BUF_633d2e19b0b0, sizeof(uint64_t) * COUNT633d2e19b0b0 * 1);
auto d_HT_633d2e19b0b0 = cuco::experimental::static_multimap{ (int)COUNT633d2e19b0b0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_633d2e13f690<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_633d2e19b0b0, d_BUF_IDX_633d2e19b0b0, d_HT_633d2e19b0b0.ref(cuco::insert), d_date__d_datekey, date_size);
//Create aggregation hash table
auto d_HT_633d2e14e9b0 = cuco::static_map{ (int)87950*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_633d2e1a9e90<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_633d2e178960, d_BUF_633d2e196490, d_BUF_633d2e19b0b0, d_BUF_633d2e19b8a0, d_HT_633d2e14e9b0.ref(cuco::insert), d_HT_633d2e178960.ref(cuco::for_each), d_HT_633d2e196490.ref(cuco::for_each), d_HT_633d2e19b0b0.ref(cuco::for_each), d_HT_633d2e19b8a0.ref(cuco::for_each), d_customer__c_nation_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
size_t COUNT633d2e14e9b0 = d_HT_633d2e14e9b0.size();
thrust::device_vector<int64_t> keys_633d2e14e9b0(COUNT633d2e14e9b0), vals_633d2e14e9b0(COUNT633d2e14e9b0);
d_HT_633d2e14e9b0.retrieve_all(keys_633d2e14e9b0.begin(), vals_633d2e14e9b0.begin());
d_HT_633d2e14e9b0.clear();
int64_t* raw_keys633d2e14e9b0 = thrust::raw_pointer_cast(keys_633d2e14e9b0.data());
insertKeys<<<std::ceil((float)COUNT633d2e14e9b0/128.), 128>>>(raw_keys633d2e14e9b0, d_HT_633d2e14e9b0.ref(cuco::insert), COUNT633d2e14e9b0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT633d2e14e9b0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT633d2e14e9b0);
DBI32Type* d_KEY_633d2e14e9b0date__d_year;
hipMalloc(&d_KEY_633d2e14e9b0date__d_year, sizeof(DBI32Type) * COUNT633d2e14e9b0);
hipMemset(d_KEY_633d2e14e9b0date__d_year, 0, sizeof(DBI32Type) * COUNT633d2e14e9b0);
DBI16Type* d_KEY_633d2e14e9b0customer__c_nation_encoded;
hipMalloc(&d_KEY_633d2e14e9b0customer__c_nation_encoded, sizeof(DBI16Type) * COUNT633d2e14e9b0);
hipMemset(d_KEY_633d2e14e9b0customer__c_nation_encoded, 0, sizeof(DBI16Type) * COUNT633d2e14e9b0);
main_633d2e1a9e90<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_633d2e178960, d_BUF_633d2e196490, d_BUF_633d2e19b0b0, d_BUF_633d2e19b8a0, d_HT_633d2e14e9b0.ref(cuco::find), d_HT_633d2e178960.ref(cuco::for_each), d_HT_633d2e196490.ref(cuco::for_each), d_HT_633d2e19b0b0.ref(cuco::for_each), d_HT_633d2e19b8a0.ref(cuco::for_each), d_KEY_633d2e14e9b0customer__c_nation_encoded, d_KEY_633d2e14e9b0date__d_year, d_aggr0__tmp_attr0, d_customer__c_nation_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, d_lineorder__lo_supplycost, lineorder_size);
//Materialize count
uint64_t* d_COUNT633d2e12db50;
hipMalloc(&d_COUNT633d2e12db50, sizeof(uint64_t));
hipMemset(d_COUNT633d2e12db50, 0, sizeof(uint64_t));
count_633d2e1bd580<<<std::ceil((float)COUNT633d2e14e9b0/128.), 128>>>(d_COUNT633d2e12db50, COUNT633d2e14e9b0);
uint64_t COUNT633d2e12db50;
hipMemcpy(&COUNT633d2e12db50, d_COUNT633d2e12db50, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX633d2e12db50;
hipMalloc(&d_MAT_IDX633d2e12db50, sizeof(uint64_t));
hipMemset(d_MAT_IDX633d2e12db50, 0, sizeof(uint64_t));
auto MAT633d2e12db50date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT633d2e12db50);
DBI32Type* d_MAT633d2e12db50date__d_year;
hipMalloc(&d_MAT633d2e12db50date__d_year, sizeof(DBI32Type) * COUNT633d2e12db50);
auto MAT633d2e12db50customer__c_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT633d2e12db50);
DBI16Type* d_MAT633d2e12db50customer__c_nation_encoded;
hipMalloc(&d_MAT633d2e12db50customer__c_nation_encoded, sizeof(DBI16Type) * COUNT633d2e12db50);
auto MAT633d2e12db50aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT633d2e12db50);
DBDecimalType* d_MAT633d2e12db50aggr0__tmp_attr0;
hipMalloc(&d_MAT633d2e12db50aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT633d2e12db50);
main_633d2e1bd580<<<std::ceil((float)COUNT633d2e14e9b0/128.), 128>>>(COUNT633d2e14e9b0, d_MAT633d2e12db50aggr0__tmp_attr0, d_MAT633d2e12db50customer__c_nation_encoded, d_MAT633d2e12db50date__d_year, d_MAT_IDX633d2e12db50, d_aggr0__tmp_attr0, d_KEY_633d2e14e9b0customer__c_nation_encoded, d_KEY_633d2e14e9b0date__d_year);
hipMemcpy(MAT633d2e12db50date__d_year, d_MAT633d2e12db50date__d_year, sizeof(DBI32Type) * COUNT633d2e12db50, hipMemcpyDeviceToHost);
hipMemcpy(MAT633d2e12db50customer__c_nation_encoded, d_MAT633d2e12db50customer__c_nation_encoded, sizeof(DBI16Type) * COUNT633d2e12db50, hipMemcpyDeviceToHost);
hipMemcpy(MAT633d2e12db50aggr0__tmp_attr0, d_MAT633d2e12db50aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT633d2e12db50, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT633d2e12db50; i++) { std::cout << "" << MAT633d2e12db50date__d_year[i];
std::cout << "," << customer__c_nation_map[MAT633d2e12db50customer__c_nation_encoded[i]];
std::cout << "," << MAT633d2e12db50aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_633d2e196490);
hipFree(d_BUF_IDX_633d2e196490);
hipFree(d_COUNT633d2e196490);
hipFree(d_BUF_633d2e178960);
hipFree(d_BUF_IDX_633d2e178960);
hipFree(d_COUNT633d2e178960);
hipFree(d_BUF_633d2e19b8a0);
hipFree(d_BUF_IDX_633d2e19b8a0);
hipFree(d_COUNT633d2e19b8a0);
hipFree(d_BUF_633d2e19b0b0);
hipFree(d_BUF_IDX_633d2e19b0b0);
hipFree(d_COUNT633d2e19b0b0);
hipFree(d_KEY_633d2e14e9b0customer__c_nation_encoded);
hipFree(d_KEY_633d2e14e9b0date__d_year);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT633d2e12db50);
hipFree(d_MAT633d2e12db50aggr0__tmp_attr0);
hipFree(d_MAT633d2e12db50customer__c_nation_encoded);
hipFree(d_MAT633d2e12db50date__d_year);
hipFree(d_MAT_IDX633d2e12db50);
free(MAT633d2e12db50aggr0__tmp_attr0);
free(MAT633d2e12db50customer__c_nation_encoded);
free(MAT633d2e12db50date__d_year);
}