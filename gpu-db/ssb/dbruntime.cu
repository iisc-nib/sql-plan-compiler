
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <dirent.h>
#include <dlfcn.h>
#include <iomanip>
#include <chrono>
#include "dbruntime.h"

int main(int argc, const char **argv)
{
    std::string dbDir = getDataDir(argv, argc);
    // std::cout << "Arg len: " << argc << std::endl;
    // assert(argc >= 4 && "Need 3 arguments, --data-dir <path to dir> <shared object query>");
    // const char *soFile = argv[3];
    initSsbDb(dbDir);

    std::cout << std::setprecision(12);

    // TODO(avinash): the runtime should listen for sql queries,
    //  compile it with the lingodb toolchain
    //  look for output.cu
    //  compile it with nvcc
    //  link it using dlopen, and execute the control function.

    // void *lib = dlopen("/media/ajayakar/space/src/sql-plan-compiler/gpu-db/tpch_auto_generated/"+ soFile, RTLD_LAZY);
    while (true)
    {

        std::string libPath;
        std::cout << "> ";
        std::cin >> libPath;
        libPath = "build/q" + libPath + ".codegen.so"; // input as integer now for convinience
        void *lib = dlopen(libPath.c_str(), RTLD_NOW);
        if (!lib)
        {
            fprintf(stderr, "%s\n", dlerror());
        }
        else
        {
            std::cout << "Opening the shared lib " << libPath << " was successful!" << std::endl;

            auto control = reinterpret_cast<void (*)(
DBI32Type*,
DBStringType*,
DBStringType*,
DBStringType*,
DBStringType*,
DBStringType*,
DBStringType*,
size_t,
DBI32Type*,
DBStringType*,
DBStringType*,
DBStringType*,
DBStringType*,
DBStringType*,
DBStringType*,
DBI32Type*,
DBStringType*,
size_t,
DBI32Type*,
DBI32Type*,
DBI32Type*,
DBI32Type*,
DBI32Type*,
DBDateType*,
DBDateType*,
DBStringType*,
DBCharType*,
DBI32Type*,
DBDecimalType*,
DBDecimalType*,
DBDecimalType*,
DBDecimalType*,
DBI32Type*,
DBI32Type*,
DBStringType*,
size_t,
DBI32Type*,
DBStringType*,
DBStringType*,
DBStringType*,
DBI32Type*,
DBI32Type*,
DBStringType*,
DBI32Type*,
DBI32Type*,
DBI32Type*,
DBI32Type*,
DBI32Type*,
DBStringType*,
DBI32Type*,
DBI32Type*,
DBI32Type*,
DBI32Type*,
size_t,
DBI32Type*,
DBStringType*,
DBStringType*,
DBStringType*,
DBStringType*,
DBStringType*,
DBStringType*,
DBStringType*,
size_t,
DBI32Type*,
DBStringType*,
DBStringType*,
size_t
               )>(dlsym(lib, "control"));
            auto start = std::chrono::high_resolution_clock::now();
            control(
 d_supplier__s_suppkey,
 d_supplier__s_name,
 d_supplier__s_address,
 d_supplier__s_city,
 d_supplier__s_nation,
 d_supplier__s_region,
 d_supplier__s_phone,
 supplier_size,
 d_part__p_partkey,
 d_part__p_name,
 d_part__p_mfgr,
 d_part__p_category,
 d_part__p_brand1,
 d_part__p_color,
 d_part__p_type,
 d_part__p_size,
 d_part__p_container,
 part_size,
 d_lineorder__lo_orderkey,
 d_lineorder__lo_linenumber,
 d_lineorder__lo_custkey,
 d_lineorder__lo_partkey,
 d_lineorder__lo_suppkey,
 d_lineorder__lo_orderdate,
 d_lineorder__lo_commitdate,
 d_lineorder__lo_orderpriority,
 d_lineorder__lo_shippriority,
 d_lineorder__lo_quantity,
 d_lineorder__lo_extendedprice,
 d_lineorder__lo_ordtotalprice,
 d_lineorder__lo_revenue,
 d_lineorder__lo_supplycost,
 d_lineorder__lo_discount,
 d_lineorder__lo_tax,
 d_lineorder__lo_shipmode,
 lineorder_size,
 d_date__d_datekey,
 d_date__d_date,
 d_date__d_dayofweek,
 d_date__d_month,
 d_date__d_year,
 d_date__d_yearmonthnum,
 d_date__d_yearmonth,
 d_date__d_daynuminweek,
 d_date__d_daynuminmonth,
 d_date__d_daynuminyear,
 d_date__d_monthnuminyear,
 d_date__d_weeknuminyear,
 d_date__d_sellingseason,
 d_date__d_lastdayinweekfl,
 d_date__d_lastdayinmonthfl,
 d_date__d_holidayfl,
 d_date__d_weekdayfl,
 date_size,
 d_customer__c_custkey,
 d_customer__c_name,
 d_customer__c_address,
 d_customer__c_city,
 d_customer__c_nation,
 d_customer__c_region,
 d_customer__c_phone,
 d_customer__c_mktsegment,
 customer_size,
 d_region__r_regionkey,
 d_region__r_name,
 d_region__r_comment,
 region_size
                );
            auto stop = std::chrono::high_resolution_clock::now();
            auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop-start);
            std::clog << "Query execution time: " << duration.count() / 1000. << "milliseconds.\n";
            dlclose(lib);
        }
    }
}
