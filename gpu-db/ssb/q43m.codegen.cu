#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5bd044c2c000(uint64_t* COUNT5bd044c23360, DBStringType* supplier__s_nation, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5bd044c23360, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5bd044c2c000(uint64_t* BUF_5bd044c23360, uint64_t* BUF_IDX_5bd044c23360, HASHTABLE_INSERT HT_5bd044c23360, DBStringType* supplier__s_nation, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_nation = supplier__s_nation[tid];
if (!(evaluatePredicate(reg_supplier__s_nation, "UNITED STATES", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5bd044c23360 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5bd044c23360 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5bd044c23360 = atomicAdd((int*)BUF_IDX_5bd044c23360, 1);
HT_5bd044c23360.insert(cuco::pair{KEY_5bd044c23360, buf_idx_5bd044c23360});
BUF_5bd044c23360[buf_idx_5bd044c23360 * 1 + 0] = tid;
}
__global__ void count_5bd044c2e9c0(uint64_t* COUNT5bd044bfd7c0, DBStringType* part__p_category, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_category = part__p_category[tid];
if (!(evaluatePredicate(reg_part__p_category, "MFGR#14", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5bd044bfd7c0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5bd044c2e9c0(uint64_t* BUF_5bd044bfd7c0, uint64_t* BUF_IDX_5bd044bfd7c0, HASHTABLE_INSERT HT_5bd044bfd7c0, DBStringType* part__p_category, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_category = part__p_category[tid];
if (!(evaluatePredicate(reg_part__p_category, "MFGR#14", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5bd044bfd7c0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_5bd044bfd7c0 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_5bd044bfd7c0 = atomicAdd((int*)BUF_IDX_5bd044bfd7c0, 1);
HT_5bd044bfd7c0.insert(cuco::pair{KEY_5bd044bfd7c0, buf_idx_5bd044bfd7c0});
BUF_5bd044bfd7c0[buf_idx_5bd044bfd7c0 * 1 + 0] = tid;
}
__global__ void count_5bd044bc3c10(uint64_t* COUNT5bd044c1b960, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5bd044c1b960, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5bd044bc3c10(uint64_t* BUF_5bd044c1b960, uint64_t* BUF_IDX_5bd044c1b960, HASHTABLE_INSERT HT_5bd044c1b960, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5bd044c1b960 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5bd044c1b960 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_5bd044c1b960 = atomicAdd((int*)BUF_IDX_5bd044c1b960, 1);
HT_5bd044c1b960.insert(cuco::pair{KEY_5bd044c1b960, buf_idx_5bd044c1b960});
BUF_5bd044c1b960[buf_idx_5bd044c1b960 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5bd044c312c0(uint64_t* BUF_5bd044bfd7c0, uint64_t* BUF_5bd044c1b960, uint64_t* BUF_5bd044c23360, uint64_t* COUNT5bd044c1b0d0, HASHTABLE_PROBE HT_5bd044bfd7c0, HASHTABLE_PROBE HT_5bd044c1b960, HASHTABLE_PROBE HT_5bd044c23360, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5bd044c23360 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5bd044c23360 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_5bd044c23360 = HT_5bd044c23360.find(KEY_5bd044c23360);
if (SLOT_5bd044c23360 == HT_5bd044c23360.end()) return;
if (!(true)) return;
uint64_t KEY_5bd044bfd7c0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5bd044bfd7c0 |= reg_lineorder__lo_partkey;
//Probe Hash table
auto SLOT_5bd044bfd7c0 = HT_5bd044bfd7c0.find(KEY_5bd044bfd7c0);
if (SLOT_5bd044bfd7c0 == HT_5bd044bfd7c0.end()) return;
if (!(true)) return;
uint64_t KEY_5bd044c1b960 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5bd044c1b960 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_5bd044c1b960 = HT_5bd044c1b960.find(KEY_5bd044c1b960);
if (SLOT_5bd044c1b960 == HT_5bd044c1b960.end()) return;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5bd044c1b0d0, 1);
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5bd044c312c0(uint64_t* BUF_5bd044bfd7c0, uint64_t* BUF_5bd044c1b0d0, uint64_t* BUF_5bd044c1b960, uint64_t* BUF_5bd044c23360, uint64_t* BUF_IDX_5bd044c1b0d0, HASHTABLE_PROBE HT_5bd044bfd7c0, HASHTABLE_INSERT HT_5bd044c1b0d0, HASHTABLE_PROBE HT_5bd044c1b960, HASHTABLE_PROBE HT_5bd044c23360, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5bd044c23360 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5bd044c23360 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_5bd044c23360 = HT_5bd044c23360.find(KEY_5bd044c23360);
if (SLOT_5bd044c23360 == HT_5bd044c23360.end()) return;
if (!(true)) return;
uint64_t KEY_5bd044bfd7c0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5bd044bfd7c0 |= reg_lineorder__lo_partkey;
//Probe Hash table
auto SLOT_5bd044bfd7c0 = HT_5bd044bfd7c0.find(KEY_5bd044bfd7c0);
if (SLOT_5bd044bfd7c0 == HT_5bd044bfd7c0.end()) return;
if (!(true)) return;
uint64_t KEY_5bd044c1b960 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5bd044c1b960 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_5bd044c1b960 = HT_5bd044c1b960.find(KEY_5bd044c1b960);
if (SLOT_5bd044c1b960 == HT_5bd044c1b960.end()) return;
if (!(true)) return;
uint64_t KEY_5bd044c1b0d0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5bd044c1b0d0 |= reg_lineorder__lo_custkey;
// Insert hash table kernel;
auto buf_idx_5bd044c1b0d0 = atomicAdd((int*)BUF_IDX_5bd044c1b0d0, 1);
HT_5bd044c1b0d0.insert(cuco::pair{KEY_5bd044c1b0d0, buf_idx_5bd044c1b0d0});
BUF_5bd044c1b0d0[buf_idx_5bd044c1b0d0 * 4 + 0] = BUF_5bd044c1b960[SLOT_5bd044c1b960->second * 1 + 0];
BUF_5bd044c1b0d0[buf_idx_5bd044c1b0d0 * 4 + 1] = BUF_5bd044c23360[SLOT_5bd044c23360->second * 1 + 0];
BUF_5bd044c1b0d0[buf_idx_5bd044c1b0d0 * 4 + 2] = BUF_5bd044bfd7c0[SLOT_5bd044bfd7c0->second * 1 + 0];
BUF_5bd044c1b0d0[buf_idx_5bd044c1b0d0 * 4 + 3] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5bd044bc39d0(uint64_t* BUF_5bd044c1b0d0, HASHTABLE_INSERT HT_5bd044bd25f0, HASHTABLE_PROBE HT_5bd044c1b0d0, DBI32Type* customer__c_custkey, size_t customer_size, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5bd044c1b0d0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5bd044c1b0d0 |= reg_customer__c_custkey;
//Probe Hash table
auto SLOT_5bd044c1b0d0 = HT_5bd044c1b0d0.find(KEY_5bd044c1b0d0);
if (SLOT_5bd044c1b0d0 == HT_5bd044c1b0d0.end()) return;
if (!(true)) return;
uint64_t KEY_5bd044bd25f0 = 0;
auto reg_date__d_year = date__d_year[BUF_5bd044c1b0d0[SLOT_5bd044c1b0d0->second * 4 + 0]];

KEY_5bd044bd25f0 |= reg_date__d_year;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_5bd044c1b0d0[SLOT_5bd044c1b0d0->second * 4 + 1]];
KEY_5bd044bd25f0 <<= 16;
KEY_5bd044bd25f0 |= reg_supplier__s_city_encoded;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_5bd044c1b0d0[SLOT_5bd044c1b0d0->second * 4 + 2]];
KEY_5bd044bd25f0 <<= 16;
KEY_5bd044bd25f0 |= reg_part__p_brand1_encoded;
//Create aggregation hash table
HT_5bd044bd25f0.insert(cuco::pair{KEY_5bd044bd25f0, 1});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5bd044bc39d0(uint64_t* BUF_5bd044c1b0d0, HASHTABLE_FIND HT_5bd044bd25f0, HASHTABLE_PROBE HT_5bd044c1b0d0, DBI32Type* KEY_5bd044bd25f0date__d_year, DBI16Type* KEY_5bd044bd25f0part__p_brand1_encoded, DBI16Type* KEY_5bd044bd25f0supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* customer__c_custkey, size_t customer_size, DBI32Type* date__d_year, DBDecimalType* lineorder__lo_revenue, DBDecimalType* lineorder__lo_supplycost, DBI16Type* part__p_brand1_encoded, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5bd044c1b0d0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5bd044c1b0d0 |= reg_customer__c_custkey;
//Probe Hash table
auto SLOT_5bd044c1b0d0 = HT_5bd044c1b0d0.find(KEY_5bd044c1b0d0);
if (SLOT_5bd044c1b0d0 == HT_5bd044c1b0d0.end()) return;
if (!(true)) return;
uint64_t KEY_5bd044bd25f0 = 0;
auto reg_date__d_year = date__d_year[BUF_5bd044c1b0d0[SLOT_5bd044c1b0d0->second * 4 + 0]];

KEY_5bd044bd25f0 |= reg_date__d_year;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_5bd044c1b0d0[SLOT_5bd044c1b0d0->second * 4 + 1]];
KEY_5bd044bd25f0 <<= 16;
KEY_5bd044bd25f0 |= reg_supplier__s_city_encoded;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_5bd044c1b0d0[SLOT_5bd044c1b0d0->second * 4 + 2]];
KEY_5bd044bd25f0 <<= 16;
KEY_5bd044bd25f0 |= reg_part__p_brand1_encoded;
//Aggregate in hashtable
auto buf_idx_5bd044bd25f0 = HT_5bd044bd25f0.find(KEY_5bd044bd25f0)->second;
auto reg_lineorder__lo_supplycost = lineorder__lo_supplycost[BUF_5bd044c1b0d0[SLOT_5bd044c1b0d0->second * 4 + 3]];
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_5bd044c1b0d0[SLOT_5bd044c1b0d0->second * 4 + 3]];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_revenue) - (reg_lineorder__lo_supplycost);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5bd044bd25f0], reg_map0__tmp_attr1);
KEY_5bd044bd25f0date__d_year[buf_idx_5bd044bd25f0] = reg_date__d_year;
KEY_5bd044bd25f0supplier__s_city_encoded[buf_idx_5bd044bd25f0] = reg_supplier__s_city_encoded;
KEY_5bd044bd25f0part__p_brand1_encoded[buf_idx_5bd044bd25f0] = reg_part__p_brand1_encoded;
}
__global__ void count_5bd044c49c60(uint64_t* COUNT5bd044bb29f0, size_t COUNT5bd044bd25f0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5bd044bd25f0) return;
//Materialize count
atomicAdd((int*)COUNT5bd044bb29f0, 1);
}
__global__ void main_5bd044c49c60(size_t COUNT5bd044bd25f0, DBDecimalType* MAT5bd044bb29f0aggr0__tmp_attr0, DBI32Type* MAT5bd044bb29f0date__d_year, DBI16Type* MAT5bd044bb29f0part__p_brand1_encoded, DBI16Type* MAT5bd044bb29f0supplier__s_city_encoded, uint64_t* MAT_IDX5bd044bb29f0, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5bd044bd25f0) return;
//Materialize buffers
auto mat_idx5bd044bb29f0 = atomicAdd((int*)MAT_IDX5bd044bb29f0, 1);
auto reg_date__d_year = date__d_year[tid];
MAT5bd044bb29f0date__d_year[mat_idx5bd044bb29f0] = reg_date__d_year;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[tid];
MAT5bd044bb29f0supplier__s_city_encoded[mat_idx5bd044bb29f0] = reg_supplier__s_city_encoded;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[tid];
MAT5bd044bb29f0part__p_brand1_encoded[mat_idx5bd044bb29f0] = reg_part__p_brand1_encoded;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5bd044bb29f0aggr0__tmp_attr0[mat_idx5bd044bb29f0] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT5bd044c23360;
hipMalloc(&d_COUNT5bd044c23360, sizeof(uint64_t));
hipMemset(d_COUNT5bd044c23360, 0, sizeof(uint64_t));
count_5bd044c2c000<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT5bd044c23360, d_supplier__s_nation, supplier_size);
uint64_t COUNT5bd044c23360;
hipMemcpy(&COUNT5bd044c23360, d_COUNT5bd044c23360, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5bd044c23360;
hipMalloc(&d_BUF_IDX_5bd044c23360, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5bd044c23360, 0, sizeof(uint64_t));
uint64_t* d_BUF_5bd044c23360;
hipMalloc(&d_BUF_5bd044c23360, sizeof(uint64_t) * COUNT5bd044c23360 * 1);
auto d_HT_5bd044c23360 = cuco::static_map{ (int)COUNT5bd044c23360*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5bd044c2c000<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_5bd044c23360, d_BUF_IDX_5bd044c23360, d_HT_5bd044c23360.ref(cuco::insert), d_supplier__s_nation, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5bd044bfd7c0;
hipMalloc(&d_COUNT5bd044bfd7c0, sizeof(uint64_t));
hipMemset(d_COUNT5bd044bfd7c0, 0, sizeof(uint64_t));
count_5bd044c2e9c0<<<std::ceil((float)part_size/128.), 128>>>(d_COUNT5bd044bfd7c0, d_part__p_category, part_size);
uint64_t COUNT5bd044bfd7c0;
hipMemcpy(&COUNT5bd044bfd7c0, d_COUNT5bd044bfd7c0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5bd044bfd7c0;
hipMalloc(&d_BUF_IDX_5bd044bfd7c0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5bd044bfd7c0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5bd044bfd7c0;
hipMalloc(&d_BUF_5bd044bfd7c0, sizeof(uint64_t) * COUNT5bd044bfd7c0 * 1);
auto d_HT_5bd044bfd7c0 = cuco::static_map{ (int)COUNT5bd044bfd7c0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5bd044c2e9c0<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_5bd044bfd7c0, d_BUF_IDX_5bd044bfd7c0, d_HT_5bd044bfd7c0.ref(cuco::insert), d_part__p_category, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT5bd044c1b960;
hipMalloc(&d_COUNT5bd044c1b960, sizeof(uint64_t));
hipMemset(d_COUNT5bd044c1b960, 0, sizeof(uint64_t));
count_5bd044bc3c10<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT5bd044c1b960, d_date__d_year, date_size);
uint64_t COUNT5bd044c1b960;
hipMemcpy(&COUNT5bd044c1b960, d_COUNT5bd044c1b960, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5bd044c1b960;
hipMalloc(&d_BUF_IDX_5bd044c1b960, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5bd044c1b960, 0, sizeof(uint64_t));
uint64_t* d_BUF_5bd044c1b960;
hipMalloc(&d_BUF_5bd044c1b960, sizeof(uint64_t) * COUNT5bd044c1b960 * 1);
auto d_HT_5bd044c1b960 = cuco::static_map{ (int)COUNT5bd044c1b960*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5bd044bc3c10<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_5bd044c1b960, d_BUF_IDX_5bd044c1b960, d_HT_5bd044c1b960.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Materialize count
uint64_t* d_COUNT5bd044c1b0d0;
hipMalloc(&d_COUNT5bd044c1b0d0, sizeof(uint64_t));
hipMemset(d_COUNT5bd044c1b0d0, 0, sizeof(uint64_t));
count_5bd044c312c0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5bd044bfd7c0, d_BUF_5bd044c1b960, d_BUF_5bd044c23360, d_COUNT5bd044c1b0d0, d_HT_5bd044bfd7c0.ref(cuco::find), d_HT_5bd044c1b960.ref(cuco::find), d_HT_5bd044c23360.ref(cuco::find), d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
uint64_t COUNT5bd044c1b0d0;
hipMemcpy(&COUNT5bd044c1b0d0, d_COUNT5bd044c1b0d0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5bd044c1b0d0;
hipMalloc(&d_BUF_IDX_5bd044c1b0d0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5bd044c1b0d0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5bd044c1b0d0;
hipMalloc(&d_BUF_5bd044c1b0d0, sizeof(uint64_t) * COUNT5bd044c1b0d0 * 4);
auto d_HT_5bd044c1b0d0 = cuco::static_map{ (int)COUNT5bd044c1b0d0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5bd044c312c0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5bd044bfd7c0, d_BUF_5bd044c1b0d0, d_BUF_5bd044c1b960, d_BUF_5bd044c23360, d_BUF_IDX_5bd044c1b0d0, d_HT_5bd044bfd7c0.ref(cuco::find), d_HT_5bd044c1b0d0.ref(cuco::insert), d_HT_5bd044c1b960.ref(cuco::find), d_HT_5bd044c23360.ref(cuco::find), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
//Create aggregation hash table
auto d_HT_5bd044bd25f0 = cuco::static_map{ (int)2259*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5bd044bc39d0<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_5bd044c1b0d0, d_HT_5bd044bd25f0.ref(cuco::insert), d_HT_5bd044c1b0d0.ref(cuco::find), d_customer__c_custkey, customer_size, d_date__d_year, d_part__p_brand1_encoded, d_supplier__s_city_encoded);
size_t COUNT5bd044bd25f0 = d_HT_5bd044bd25f0.size();
thrust::device_vector<int64_t> keys_5bd044bd25f0(COUNT5bd044bd25f0), vals_5bd044bd25f0(COUNT5bd044bd25f0);
d_HT_5bd044bd25f0.retrieve_all(keys_5bd044bd25f0.begin(), vals_5bd044bd25f0.begin());
d_HT_5bd044bd25f0.clear();
int64_t* raw_keys5bd044bd25f0 = thrust::raw_pointer_cast(keys_5bd044bd25f0.data());
insertKeys<<<std::ceil((float)COUNT5bd044bd25f0/128.), 128>>>(raw_keys5bd044bd25f0, d_HT_5bd044bd25f0.ref(cuco::insert), COUNT5bd044bd25f0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5bd044bd25f0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5bd044bd25f0);
DBI32Type* d_KEY_5bd044bd25f0date__d_year;
hipMalloc(&d_KEY_5bd044bd25f0date__d_year, sizeof(DBI32Type) * COUNT5bd044bd25f0);
hipMemset(d_KEY_5bd044bd25f0date__d_year, 0, sizeof(DBI32Type) * COUNT5bd044bd25f0);
DBI16Type* d_KEY_5bd044bd25f0supplier__s_city_encoded;
hipMalloc(&d_KEY_5bd044bd25f0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5bd044bd25f0);
hipMemset(d_KEY_5bd044bd25f0supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT5bd044bd25f0);
DBI16Type* d_KEY_5bd044bd25f0part__p_brand1_encoded;
hipMalloc(&d_KEY_5bd044bd25f0part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5bd044bd25f0);
hipMemset(d_KEY_5bd044bd25f0part__p_brand1_encoded, 0, sizeof(DBI16Type) * COUNT5bd044bd25f0);
main_5bd044bc39d0<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_5bd044c1b0d0, d_HT_5bd044bd25f0.ref(cuco::find), d_HT_5bd044c1b0d0.ref(cuco::find), d_KEY_5bd044bd25f0date__d_year, d_KEY_5bd044bd25f0part__p_brand1_encoded, d_KEY_5bd044bd25f0supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_custkey, customer_size, d_date__d_year, d_lineorder__lo_revenue, d_lineorder__lo_supplycost, d_part__p_brand1_encoded, d_supplier__s_city_encoded);
//Materialize count
uint64_t* d_COUNT5bd044bb29f0;
hipMalloc(&d_COUNT5bd044bb29f0, sizeof(uint64_t));
hipMemset(d_COUNT5bd044bb29f0, 0, sizeof(uint64_t));
count_5bd044c49c60<<<std::ceil((float)COUNT5bd044bd25f0/128.), 128>>>(d_COUNT5bd044bb29f0, COUNT5bd044bd25f0);
uint64_t COUNT5bd044bb29f0;
hipMemcpy(&COUNT5bd044bb29f0, d_COUNT5bd044bb29f0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5bd044bb29f0;
hipMalloc(&d_MAT_IDX5bd044bb29f0, sizeof(uint64_t));
hipMemset(d_MAT_IDX5bd044bb29f0, 0, sizeof(uint64_t));
auto MAT5bd044bb29f0date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5bd044bb29f0);
DBI32Type* d_MAT5bd044bb29f0date__d_year;
hipMalloc(&d_MAT5bd044bb29f0date__d_year, sizeof(DBI32Type) * COUNT5bd044bb29f0);
auto MAT5bd044bb29f0supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5bd044bb29f0);
DBI16Type* d_MAT5bd044bb29f0supplier__s_city_encoded;
hipMalloc(&d_MAT5bd044bb29f0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5bd044bb29f0);
auto MAT5bd044bb29f0part__p_brand1_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5bd044bb29f0);
DBI16Type* d_MAT5bd044bb29f0part__p_brand1_encoded;
hipMalloc(&d_MAT5bd044bb29f0part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5bd044bb29f0);
auto MAT5bd044bb29f0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5bd044bb29f0);
DBDecimalType* d_MAT5bd044bb29f0aggr0__tmp_attr0;
hipMalloc(&d_MAT5bd044bb29f0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5bd044bb29f0);
main_5bd044c49c60<<<std::ceil((float)COUNT5bd044bd25f0/128.), 128>>>(COUNT5bd044bd25f0, d_MAT5bd044bb29f0aggr0__tmp_attr0, d_MAT5bd044bb29f0date__d_year, d_MAT5bd044bb29f0part__p_brand1_encoded, d_MAT5bd044bb29f0supplier__s_city_encoded, d_MAT_IDX5bd044bb29f0, d_aggr0__tmp_attr0, d_KEY_5bd044bd25f0date__d_year, d_KEY_5bd044bd25f0part__p_brand1_encoded, d_KEY_5bd044bd25f0supplier__s_city_encoded);
hipMemcpy(MAT5bd044bb29f0date__d_year, d_MAT5bd044bb29f0date__d_year, sizeof(DBI32Type) * COUNT5bd044bb29f0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5bd044bb29f0supplier__s_city_encoded, d_MAT5bd044bb29f0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT5bd044bb29f0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5bd044bb29f0part__p_brand1_encoded, d_MAT5bd044bb29f0part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5bd044bb29f0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5bd044bb29f0aggr0__tmp_attr0, d_MAT5bd044bb29f0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5bd044bb29f0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5bd044bb29f0; i++) { std::cout << "" << MAT5bd044bb29f0date__d_year[i];
std::cout << "," << supplier__s_city_map[MAT5bd044bb29f0supplier__s_city_encoded[i]];
std::cout << "," << part__p_brand1_map[MAT5bd044bb29f0part__p_brand1_encoded[i]];
std::cout << "," << MAT5bd044bb29f0aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_5bd044c23360);
hipFree(d_BUF_IDX_5bd044c23360);
hipFree(d_COUNT5bd044c23360);
hipFree(d_BUF_5bd044bfd7c0);
hipFree(d_BUF_IDX_5bd044bfd7c0);
hipFree(d_COUNT5bd044bfd7c0);
hipFree(d_BUF_5bd044c1b960);
hipFree(d_BUF_IDX_5bd044c1b960);
hipFree(d_COUNT5bd044c1b960);
hipFree(d_BUF_5bd044c1b0d0);
hipFree(d_BUF_IDX_5bd044c1b0d0);
hipFree(d_COUNT5bd044c1b0d0);
hipFree(d_KEY_5bd044bd25f0date__d_year);
hipFree(d_KEY_5bd044bd25f0part__p_brand1_encoded);
hipFree(d_KEY_5bd044bd25f0supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5bd044bb29f0);
hipFree(d_MAT5bd044bb29f0aggr0__tmp_attr0);
hipFree(d_MAT5bd044bb29f0date__d_year);
hipFree(d_MAT5bd044bb29f0part__p_brand1_encoded);
hipFree(d_MAT5bd044bb29f0supplier__s_city_encoded);
hipFree(d_MAT_IDX5bd044bb29f0);
free(MAT5bd044bb29f0aggr0__tmp_attr0);
free(MAT5bd044bb29f0date__d_year);
free(MAT5bd044bb29f0part__p_brand1_encoded);
free(MAT5bd044bb29f0supplier__s_city_encoded);
}