#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBI32Type* customer__c_custkey, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0, buf_idx_0});
BUF_0[buf_idx_0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_INSERT HT_2, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_2 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_2 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_2 = atomicAdd((int*)BUF_IDX_2, 1);
HT_2.insert(cuco::pair{KEY_2, buf_idx_2});
BUF_2[buf_idx_2 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_INSERT HT_4, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_4 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_4 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_4 = atomicAdd((int*)BUF_IDX_4, 1);
HT_4.insert(cuco::pair{KEY_4, buf_idx_4});
BUF_4[buf_idx_4 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT>
__global__ void main_7(uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_INSERT HT_6, DBStringType* part__p_mfgr, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_mfgr = part__p_mfgr[tid];
if (!((evaluatePredicate(reg_part__p_mfgr, "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr, "MFGR#2", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_6 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_6 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_6 = atomicAdd((int*)BUF_IDX_6, 1);
HT_6.insert(cuco::pair{KEY_6, buf_idx_6});
BUF_6[buf_idx_6 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_FIND>
__global__ void main_9(uint64_t* BUF_0, uint64_t* BUF_2, uint64_t* BUF_4, uint64_t* BUF_6, HASHTABLE_PROBE_PK HT_0, HASHTABLE_PROBE_PK HT_2, HASHTABLE_PROBE_PK HT_4, HASHTABLE_PROBE_PK HT_6, HASHTABLE_FIND HT_8, DBI32Type* KEY_8date__d_year, DBI16Type* KEY_8part__p_category_encoded, DBI16Type* KEY_8supplier__s_nation_encoded, int* SLOT_COUNT_8, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, DBDecimalType* lineorder__lo_supplycost, size_t lineorder_size, DBI16Type* part__p_category_encoded, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_0 |= reg_lineorder__lo_custkey;
//Probe Hash table
auto SLOT_0 = HT_0.find(KEY_0);
if (SLOT_0 == HT_0.end()) return;
if (!(true)) return;
uint64_t KEY_2 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_2 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_2 = HT_2.find(KEY_2);
if (SLOT_2 == HT_2.end()) return;
if (!(true)) return;
uint64_t KEY_4 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_4 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_4 = HT_4.find(KEY_4);
if (SLOT_4 == HT_4.end()) return;
if (!(true)) return;
uint64_t KEY_6 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_6 |= reg_lineorder__lo_partkey;
//Probe Hash table
auto SLOT_6 = HT_6.find(KEY_6);
if (SLOT_6 == HT_6.end()) return;
if (!(true)) return;
uint64_t KEY_8 = 0;
auto reg_date__d_year = date__d_year[BUF_4[SLOT_4->second * 1 + 0]];

KEY_8 |= reg_date__d_year;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_2[SLOT_2->second * 1 + 0]];
KEY_8 <<= 16;
KEY_8 |= reg_supplier__s_nation_encoded;
auto reg_part__p_category_encoded = part__p_category_encoded[BUF_6[SLOT_6->second * 1 + 0]];
KEY_8 <<= 16;
KEY_8 |= reg_part__p_category_encoded;
//Aggregate in hashtable
auto buf_idx_8 = get_aggregation_slot(KEY_8, HT_8, SLOT_COUNT_8);
auto reg_lineorder__lo_supplycost = lineorder__lo_supplycost[tid];
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_revenue) - (reg_lineorder__lo_supplycost);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_8], reg_map0__tmp_attr1);
KEY_8date__d_year[buf_idx_8] = reg_date__d_year;
KEY_8supplier__s_nation_encoded[buf_idx_8] = reg_supplier__s_nation_encoded;
KEY_8part__p_category_encoded[buf_idx_8] = reg_part__p_category_encoded;
}
__global__ void main_11(size_t COUNT8, DBDecimalType* MAT10aggr0__tmp_attr0, DBI32Type* MAT10date__d_year, DBI16Type* MAT10part__p_category_encoded, DBI16Type* MAT10supplier__s_nation_encoded, uint64_t* MAT_IDX10, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_category_encoded, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT8) return;
//Materialize buffers
auto mat_idx10 = atomicAdd((int*)MAT_IDX10, 1);
auto reg_date__d_year = date__d_year[tid];
MAT10date__d_year[mat_idx10] = reg_date__d_year;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[tid];
MAT10supplier__s_nation_encoded[mat_idx10] = reg_supplier__s_nation_encoded;
auto reg_part__p_category_encoded = part__p_category_encoded[tid];
MAT10part__p_category_encoded[mat_idx10] = reg_part__p_category_encoded;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT10aggr0__tmp_attr0[mat_idx10] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
size_t COUNT0 = customer_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_customer__c_custkey, d_customer__c_region, customer_size);
hipFree(d_BUF_IDX_0);
size_t COUNT2 = supplier_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 1);
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_2, d_BUF_IDX_2, d_HT_2.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
hipFree(d_BUF_IDX_2);
size_t COUNT4 = date_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 1);
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_4, d_BUF_IDX_4, d_HT_4.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
hipFree(d_BUF_IDX_4);
size_t COUNT6 = part_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 1);
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_6, d_BUF_IDX_6, d_HT_6.ref(cuco::insert), d_part__p_mfgr, d_part__p_partkey, part_size);
hipFree(d_BUF_IDX_6);
size_t COUNT8 = 257987;
auto d_HT_8 = cuco::static_map{ (int)257987*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_8;
hipMalloc(&d_SLOT_COUNT_8, sizeof(int));
hipMemset(d_SLOT_COUNT_8, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT8);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT8);
DBI32Type* d_KEY_8date__d_year;
hipMalloc(&d_KEY_8date__d_year, sizeof(DBI32Type) * COUNT8);
hipMemset(d_KEY_8date__d_year, 0, sizeof(DBI32Type) * COUNT8);
DBI16Type* d_KEY_8supplier__s_nation_encoded;
hipMalloc(&d_KEY_8supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT8);
hipMemset(d_KEY_8supplier__s_nation_encoded, 0, sizeof(DBI16Type) * COUNT8);
DBI16Type* d_KEY_8part__p_category_encoded;
hipMalloc(&d_KEY_8part__p_category_encoded, sizeof(DBI16Type) * COUNT8);
hipMemset(d_KEY_8part__p_category_encoded, 0, sizeof(DBI16Type) * COUNT8);
main_9<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_0, d_BUF_2, d_BUF_4, d_BUF_6, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::insert_and_find), d_KEY_8date__d_year, d_KEY_8part__p_category_encoded, d_KEY_8supplier__s_nation_encoded, d_SLOT_COUNT_8, d_aggr0__tmp_attr0, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, d_lineorder__lo_supplycost, lineorder_size, d_part__p_category_encoded, d_supplier__s_nation_encoded);
COUNT8 = d_HT_8.size();
size_t COUNT10 = COUNT8;
//Materialize buffers
uint64_t* d_MAT_IDX10;
hipMalloc(&d_MAT_IDX10, sizeof(uint64_t));
hipMemset(d_MAT_IDX10, 0, sizeof(uint64_t));
auto MAT10date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT10);
DBI32Type* d_MAT10date__d_year;
hipMalloc(&d_MAT10date__d_year, sizeof(DBI32Type) * COUNT10);
auto MAT10supplier__s_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT10);
DBI16Type* d_MAT10supplier__s_nation_encoded;
hipMalloc(&d_MAT10supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT10);
auto MAT10part__p_category_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT10);
DBI16Type* d_MAT10part__p_category_encoded;
hipMalloc(&d_MAT10part__p_category_encoded, sizeof(DBI16Type) * COUNT10);
auto MAT10aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT10);
DBDecimalType* d_MAT10aggr0__tmp_attr0;
hipMalloc(&d_MAT10aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT10);
main_11<<<std::ceil((float)COUNT8/128.), 128>>>(COUNT8, d_MAT10aggr0__tmp_attr0, d_MAT10date__d_year, d_MAT10part__p_category_encoded, d_MAT10supplier__s_nation_encoded, d_MAT_IDX10, d_aggr0__tmp_attr0, d_KEY_8date__d_year, d_KEY_8part__p_category_encoded, d_KEY_8supplier__s_nation_encoded);
uint64_t MATCOUNT_10 = 0;
hipMemcpy(&MATCOUNT_10, d_MAT_IDX10, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT10date__d_year, d_MAT10date__d_year, sizeof(DBI32Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10supplier__s_nation_encoded, d_MAT10supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10part__p_category_encoded, d_MAT10part__p_category_encoded, sizeof(DBI16Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10aggr0__tmp_attr0, d_MAT10aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT10, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < MATCOUNT_10; i++) { std::cout << "" << MAT10date__d_year[i];
std::cout << "|" << supplier__s_nation_map[MAT10supplier__s_nation_encoded[i]];
std::cout << "|" << part__p_category_map[MAT10part__p_category_encoded[i]];
std::cout << "|" << MAT10aggr0__tmp_attr0[i];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_KEY_8date__d_year);
hipFree(d_KEY_8part__p_category_encoded);
hipFree(d_KEY_8supplier__s_nation_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_MAT10aggr0__tmp_attr0);
hipFree(d_MAT10date__d_year);
hipFree(d_MAT10part__p_category_encoded);
hipFree(d_MAT10supplier__s_nation_encoded);
hipFree(d_MAT_IDX10);
free(MAT10aggr0__tmp_attr0);
free(MAT10date__d_year);
free(MAT10part__p_category_encoded);
free(MAT10supplier__s_nation_encoded);
}