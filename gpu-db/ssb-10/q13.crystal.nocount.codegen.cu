#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBI32Type* date__d_datekey, DBI32Type* date__d_weeknuminyear, DBI32Type* date__d_year, size_t date_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBI32Type reg_date__d_weeknuminyear[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_weeknuminyear[ITEM] = date__d_weeknuminyear[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_date__d_weeknuminyear[ITEM], 6, Predicate::eq);
}
DBI32Type reg_date__d_year[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_year[ITEM] = date__d_year[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_date__d_year[ITEM], 1994, Predicate::eq);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_date__d_datekey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
reg_date__d_datekey[ITEM] = date__d_datekey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_date__d_datekey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < date_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_0.insert(cuco::pair{KEY_0[ITEM], ITEM*TB + tid});
BUF_0[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_3(uint64_t* BUF_0, HASHTABLE_PROBE HT_0, HASHTABLE_FIND HT_2, int* SLOT_COUNT_2, DBDecimalType* aggr0__tmp_attr0, DBI32Type* lineorder__lo_discount, DBDecimalType* lineorder__lo_extendedprice, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBI32Type reg_lineorder__lo_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_discount[ITEM] = lineorder__lo_discount[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineorder__lo_discount[ITEM], 5, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount[ITEM], 7, Predicate::lte);
}
DBI32Type reg_lineorder__lo_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_quantity[ITEM] = lineorder__lo_quantity[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineorder__lo_quantity[ITEM], 26, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_quantity[ITEM], 35, Predicate::lte);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_lineorder__lo_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_orderdate[ITEM] = lineorder__lo_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_lineorder__lo_orderdate[ITEM];
}
//Probe Hash table
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
}
//Aggregate in hashtable
DBDecimalType reg_lineorder__lo_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
reg_lineorder__lo_extendedprice[ITEM] = lineorder__lo_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineorder__lo_extendedprice[ITEM]) * ((DBDecimalType)(reg_lineorder__lo_discount[ITEM]));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineorder_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_2 = get_aggregation_slot(KEY_2[ITEM], HT_2, SLOT_COUNT_2);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_2], reg_map0__tmp_attr1[ITEM]);
}
}
__global__ void main_5(size_t COUNT2, DBDecimalType* MAT4aggr0__tmp_attr0, uint64_t* MAT_IDX4, DBDecimalType* aggr0__tmp_attr0) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize buffers
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx4 = atomicAdd((int*)MAT_IDX4, 1);
MAT4aggr0__tmp_attr0[mat_idx4] = reg_aggr0__tmp_attr0[ITEM];
}
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
size_t COUNT0 = date_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)date_size/(float)TILE_SIZE), TB>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_date__d_datekey, d_date__d_weeknuminyear, d_date__d_year, date_size);
size_t COUNT2 = 1;
auto d_HT_2 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_2;
hipMalloc(&d_SLOT_COUNT_2, sizeof(int));
hipMemset(d_SLOT_COUNT_2, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT2);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT2);
main_3<<<std::ceil((float)lineorder_size/(float)TILE_SIZE), TB>>>(d_BUF_0, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::insert_and_find), d_SLOT_COUNT_2, d_aggr0__tmp_attr0, d_lineorder__lo_discount, d_lineorder__lo_extendedprice, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
COUNT2 = d_HT_2.size();
size_t COUNT4 = COUNT2;
//Materialize buffers
uint64_t* d_MAT_IDX4;
hipMalloc(&d_MAT_IDX4, sizeof(uint64_t));
hipMemset(d_MAT_IDX4, 0, sizeof(uint64_t));
auto MAT4aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT4);
DBDecimalType* d_MAT4aggr0__tmp_attr0;
hipMalloc(&d_MAT4aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT4);
main_5<<<std::ceil((float)COUNT2/(float)TILE_SIZE), TB>>>(COUNT2, d_MAT4aggr0__tmp_attr0, d_MAT_IDX4, d_aggr0__tmp_attr0);
uint64_t MATCOUNT_4 = 0;
hipMemcpy(&MATCOUNT_4, d_MAT_IDX4, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT4aggr0__tmp_attr0, d_MAT4aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT4, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < MATCOUNT_4; i++) { std::cout << "" << MAT4aggr0__tmp_attr0[i];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_MAT4aggr0__tmp_attr0);
hipFree(d_MAT_IDX4);
free(MAT4aggr0__tmp_attr0);
}