#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBStringType* part__p_category, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_category = part__p_category[tid];
if (!(evaluatePredicate(reg_part__p_category, "MFGR#12", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_0 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0, buf_idx_0});
BUF_0[buf_idx_0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_INSERT HT_2, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_2 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_2 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_2 = atomicAdd((int*)BUF_IDX_2, 1);
HT_2.insert(cuco::pair{KEY_2, buf_idx_2});
BUF_2[buf_idx_2 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_INSERT HT_4, DBI32Type* date__d_datekey, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_4 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_4 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_4 = atomicAdd((int*)BUF_IDX_4, 1);
HT_4.insert(cuco::pair{KEY_4, buf_idx_4});
BUF_4[buf_idx_4 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_FIND>
__global__ void main_7(uint64_t* BUF_0, uint64_t* BUF_2, uint64_t* BUF_4, HASHTABLE_PROBE_PK HT_0, HASHTABLE_PROBE_PK HT_2, HASHTABLE_PROBE_PK HT_4, HASHTABLE_FIND HT_6, DBI32Type* KEY_6date__d_year, DBI16Type* KEY_6part__p_brand1_encoded, int* SLOT_COUNT_6, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_0 |= reg_lineorder__lo_partkey;
//Probe Hash table
auto SLOT_0 = HT_0.find(KEY_0);
if (SLOT_0 == HT_0.end()) return;
if (!(true)) return;
uint64_t KEY_2 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_2 |= reg_lineorder__lo_suppkey;
//Probe Hash table
auto SLOT_2 = HT_2.find(KEY_2);
if (SLOT_2 == HT_2.end()) return;
if (!(true)) return;
uint64_t KEY_4 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_4 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_4 = HT_4.find(KEY_4);
if (SLOT_4 == HT_4.end()) return;
if (!(true)) return;
uint64_t KEY_6 = 0;
auto reg_date__d_year = date__d_year[BUF_4[SLOT_4->second * 1 + 0]];

KEY_6 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_0[SLOT_0->second * 1 + 0]];
KEY_6 <<= 16;
KEY_6 |= reg_part__p_brand1_encoded;
//Aggregate in hashtable
auto buf_idx_6 = get_aggregation_slot(KEY_6, HT_6, SLOT_COUNT_6);
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_6], reg_lineorder__lo_revenue);
KEY_6date__d_year[buf_idx_6] = reg_date__d_year;
KEY_6part__p_brand1_encoded[buf_idx_6] = reg_part__p_brand1_encoded;
}
__global__ void main_9(size_t COUNT6, DBDecimalType* MAT8aggr0__tmp_attr0, DBI32Type* MAT8date__d_year, DBI16Type* MAT8part__p_brand1_encoded, uint64_t* MAT_IDX8, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT6) return;
//Materialize buffers
auto mat_idx8 = atomicAdd((int*)MAT_IDX8, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT8aggr0__tmp_attr0[mat_idx8] = reg_aggr0__tmp_attr0;
auto reg_date__d_year = date__d_year[tid];
MAT8date__d_year[mat_idx8] = reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[tid];
MAT8part__p_brand1_encoded[mat_idx8] = reg_part__p_brand1_encoded;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
size_t COUNT0 = part_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_part__p_category, d_part__p_partkey, part_size);
hipFree(d_BUF_IDX_0);
size_t COUNT2 = supplier_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 1);
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_2, d_BUF_IDX_2, d_HT_2.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
hipFree(d_BUF_IDX_2);
size_t COUNT4 = date_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 1);
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_4, d_BUF_IDX_4, d_HT_4.ref(cuco::insert), d_date__d_datekey, date_size);
hipFree(d_BUF_IDX_4);
size_t COUNT6 = 391755;
auto d_HT_6 = cuco::static_map{ (int)391755*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_6;
hipMalloc(&d_SLOT_COUNT_6, sizeof(int));
hipMemset(d_SLOT_COUNT_6, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT6);
DBI32Type* d_KEY_6date__d_year;
hipMalloc(&d_KEY_6date__d_year, sizeof(DBI32Type) * COUNT6);
hipMemset(d_KEY_6date__d_year, 0, sizeof(DBI32Type) * COUNT6);
DBI16Type* d_KEY_6part__p_brand1_encoded;
hipMalloc(&d_KEY_6part__p_brand1_encoded, sizeof(DBI16Type) * COUNT6);
hipMemset(d_KEY_6part__p_brand1_encoded, 0, sizeof(DBI16Type) * COUNT6);
main_7<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_0, d_BUF_2, d_BUF_4, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::insert_and_find), d_KEY_6date__d_year, d_KEY_6part__p_brand1_encoded, d_SLOT_COUNT_6, d_aggr0__tmp_attr0, d_date__d_year, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_part__p_brand1_encoded);
COUNT6 = d_HT_6.size();
size_t COUNT8 = COUNT6;
//Materialize buffers
uint64_t* d_MAT_IDX8;
hipMalloc(&d_MAT_IDX8, sizeof(uint64_t));
hipMemset(d_MAT_IDX8, 0, sizeof(uint64_t));
auto MAT8aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT8);
DBDecimalType* d_MAT8aggr0__tmp_attr0;
hipMalloc(&d_MAT8aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT8);
auto MAT8date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT8);
DBI32Type* d_MAT8date__d_year;
hipMalloc(&d_MAT8date__d_year, sizeof(DBI32Type) * COUNT8);
auto MAT8part__p_brand1_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT8);
DBI16Type* d_MAT8part__p_brand1_encoded;
hipMalloc(&d_MAT8part__p_brand1_encoded, sizeof(DBI16Type) * COUNT8);
main_9<<<std::ceil((float)COUNT6/128.), 128>>>(COUNT6, d_MAT8aggr0__tmp_attr0, d_MAT8date__d_year, d_MAT8part__p_brand1_encoded, d_MAT_IDX8, d_aggr0__tmp_attr0, d_KEY_6date__d_year, d_KEY_6part__p_brand1_encoded);
uint64_t MATCOUNT_8 = 0;
hipMemcpy(&MATCOUNT_8, d_MAT_IDX8, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT8aggr0__tmp_attr0, d_MAT8aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT8, hipMemcpyDeviceToHost);
hipMemcpy(MAT8date__d_year, d_MAT8date__d_year, sizeof(DBI32Type) * COUNT8, hipMemcpyDeviceToHost);
hipMemcpy(MAT8part__p_brand1_encoded, d_MAT8part__p_brand1_encoded, sizeof(DBI16Type) * COUNT8, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < MATCOUNT_8; i++) { std::cout << "" << MAT8aggr0__tmp_attr0[i];
std::cout << "|" << MAT8date__d_year[i];
std::cout << "|" << part__p_brand1_map[MAT8part__p_brand1_encoded[i]];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_KEY_6date__d_year);
hipFree(d_KEY_6part__p_brand1_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_MAT8aggr0__tmp_attr0);
hipFree(d_MAT8date__d_year);
hipFree(d_MAT8part__p_brand1_encoded);
hipFree(d_MAT_IDX8);
free(MAT8aggr0__tmp_attr0);
free(MAT8date__d_year);
free(MAT8part__p_brand1_encoded);
}