#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_FIND>
__global__ void main_1(HASHTABLE_FIND HT_0, DBI32Type* KEY_0lineitem_u_1__l_orderkey, int* SLOT_COUNT_0, DBDecimalType* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_1_aggregation_0, size_t lineitem_size, DBI32Type* lineitem_u_1__l_orderkey, DBDecimalType* lineitem_u_1__l_quantity) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_lineitem_u_1__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem_u_1__l_orderkey[ITEM] = lineitem_u_1__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_lineitem_u_1__l_orderkey[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem_u_1__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem_u_1__l_quantity[ITEM] = lineitem_u_1__l_quantity[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_0 = get_aggregation_slot(KEY_0[ITEM], HT_0, SLOT_COUNT_0);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_0], reg_lineitem_u_1__l_quantity[ITEM]);
KEY_0lineitem_u_1__l_orderkey[buf_idx_0] = reg_lineitem_u_1__l_orderkey[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_aggregation_0[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT_SJ>
__global__ void main_3(size_t COUNT0, HASHTABLE_INSERT_SJ HT_4, DBDecimalType* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_3_selection_2, int64_t* cycles_per_warp_main_3_semi_join_build_4, DBI32Type* lineitem_u_1__l_orderkey) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_aggr0__tmp_attr0[ITEM], 300.0, Predicate::gt);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_lineitem_u_1__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_lineitem_u_1__l_orderkey[ITEM] = lineitem_u_1__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_lineitem_u_1__l_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_4.insert(cuco::pair{KEY_4[ITEM], 1});
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_semi_join_build_4[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT>
__global__ void main_7(uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_INSERT HT_6, DBI32Type* customer__c_custkey, size_t customer_size, int64_t* cycles_per_warp_main_7_join_build_6) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_6.insert(cuco::pair{KEY_6[ITEM], ITEM*TB + tid});
BUF_6[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_build_6[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE_SJ, typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_6, uint64_t* BUF_8, uint64_t* BUF_IDX_8, HASHTABLE_PROBE_SJ HT_4, HASHTABLE_PROBE HT_6, HASHTABLE_INSERT HT_8, int64_t* cycles_per_warp_main_5_join_build_8, int64_t* cycles_per_warp_main_5_join_probe_6, int64_t* cycles_per_warp_main_5_semi_join_probe_4, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_orders__o_orderkey[ITEM];
}
//Probe Hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0;}
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_semi_join_probe_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_custkey[ITEM] = orders__o_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_orders__o_custkey[ITEM];
}
//Probe Hash table
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_join_probe_6[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_8[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_orders__o_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_8.insert(cuco::pair{KEY_8[ITEM], ITEM*TB + tid});
BUF_8[(ITEM*TB + tid) * 2 + 0] = BUF_6[slot_second6[ITEM] * 1 + 0];
BUF_8[(ITEM*TB + tid) * 2 + 1] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_join_build_8[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_9(uint64_t* BUF_8, HASHTABLE_FIND HT_10, HASHTABLE_PROBE HT_8, DBI32Type* KEY_10orders__o_orderkey, int* SLOT_COUNT_10, DBDecimalType* aggr1__tmp_attr1, DBI32Type* aggr__c_custkey, DBDateType* aggr__o_orderdate, DBDecimalType* aggr__o_totalprice, DBI32Type* customer__c_custkey, int64_t* cycles_per_warp_main_9_aggregation_10, int64_t* cycles_per_warp_main_9_join_probe_8, DBI32Type* lineitem__l_orderkey, DBDecimalType* lineitem__l_quantity, size_t lineitem_size, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, DBDecimalType* orders__o_totalprice) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
//Probe Hash table
int64_t slot_second8[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_8 = HT_8.find(KEY_8[ITEM]);
if (SLOT_8 == HT_8.end()) {selection_flags[ITEM] = 0; continue;}
slot_second8[ITEM] = SLOT_8->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_9_join_probe_8[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[BUF_8[slot_second8[ITEM] * 2 + 1]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_orders__o_orderkey[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_quantity[ITEM] = lineitem__l_quantity[ITEM*TB + tid];
}
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_custkey[ITEM] = customer__c_custkey[BUF_8[slot_second8[ITEM] * 2 + 0]];
}
DBDecimalType reg_orders__o_totalprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_totalprice[ITEM] = orders__o_totalprice[BUF_8[slot_second8[ITEM] * 2 + 1]];
}
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[BUF_8[slot_second8[ITEM] * 2 + 1]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_10 = get_aggregation_slot(KEY_10[ITEM], HT_10, SLOT_COUNT_10);
aggregate_sum(&aggr1__tmp_attr1[buf_idx_10], reg_lineitem__l_quantity[ITEM]);
aggregate_any(&aggr__c_custkey[buf_idx_10], reg_customer__c_custkey[ITEM]);
aggregate_any(&aggr__o_totalprice[buf_idx_10], reg_orders__o_totalprice[ITEM]);
aggregate_any(&aggr__o_orderdate[buf_idx_10], reg_orders__o_orderdate[ITEM]);
KEY_10orders__o_orderkey[buf_idx_10] = reg_orders__o_orderkey[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_9_aggregation_10[blockIdx.x] = cycles_per_warp;}
}
__global__ void main_12(size_t COUNT10, DBDecimalType* MAT11aggr1__tmp_attr1, DBI32Type* MAT11aggr__c_custkey, DBDateType* MAT11aggr__o_orderdate, DBDecimalType* MAT11aggr__o_totalprice, DBI32Type* MAT11orders__o_orderkey, uint64_t* MAT_IDX11, DBDecimalType* aggr1__tmp_attr1, DBI32Type* aggr__c_custkey, DBDateType* aggr__o_orderdate, DBDecimalType* aggr__o_totalprice, int64_t* cycles_per_warp_main_12_materialize_11, DBI32Type* orders__o_orderkey) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBI32Type reg_aggr__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_aggr__c_custkey[ITEM] = aggr__c_custkey[ITEM*TB + tid];
}
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
DBDateType reg_aggr__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_aggr__o_orderdate[ITEM] = aggr__o_orderdate[ITEM*TB + tid];
}
DBDecimalType reg_aggr__o_totalprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_aggr__o_totalprice[ITEM] = aggr__o_totalprice[ITEM*TB + tid];
}
DBDecimalType reg_aggr1__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_aggr1__tmp_attr1[ITEM] = aggr1__tmp_attr1[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx11 = atomicAdd((int*)MAT_IDX11, 1);
MAT11aggr__c_custkey[mat_idx11] = reg_aggr__c_custkey[ITEM];
MAT11orders__o_orderkey[mat_idx11] = reg_orders__o_orderkey[ITEM];
MAT11aggr__o_orderdate[mat_idx11] = reg_aggr__o_orderdate[ITEM];
MAT11aggr__o_totalprice[mat_idx11] = reg_aggr__o_totalprice[ITEM];
MAT11aggr1__tmp_attr1[mat_idx11] = reg_aggr1__tmp_attr1[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_12_materialize_11[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_1_aggregation_0;
auto main_1_aggregation_0_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_aggregation_0, sizeof(int64_t) * main_1_aggregation_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_aggregation_0, -1, sizeof(int64_t) * main_1_aggregation_0_cpw_size);
size_t COUNT0 = 6001215;
auto d_HT_0 = cuco::static_map{ (int)6001215*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_0;
hipMalloc(&d_SLOT_COUNT_0, sizeof(int));
hipMemset(d_SLOT_COUNT_0, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT0);
DBI32Type* d_KEY_0lineitem_u_1__l_orderkey;
hipMalloc(&d_KEY_0lineitem_u_1__l_orderkey, sizeof(DBI32Type) * COUNT0);
hipMemset(d_KEY_0lineitem_u_1__l_orderkey, 0, sizeof(DBI32Type) * COUNT0);
main_1<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TB>>>(d_HT_0.ref(cuco::insert_and_find), d_KEY_0lineitem_u_1__l_orderkey, d_SLOT_COUNT_0, d_aggr0__tmp_attr0, d_cycles_per_warp_main_1_aggregation_0, lineitem_size, d_lineitem__l_orderkey, d_lineitem__l_quantity);
int64_t* cycles_per_warp_main_1_aggregation_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_aggregation_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_aggregation_0, d_cycles_per_warp_main_1_aggregation_0, sizeof(int64_t) * main_1_aggregation_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_aggregation_0 ";
for (auto i=0ull; i < main_1_aggregation_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_aggregation_0[i] << " ";
std::cout << std::endl;
COUNT0 = d_HT_0.size();
int64_t* d_cycles_per_warp_main_3_selection_2;
auto main_3_selection_2_cpw_size = std::ceil((float)COUNT0/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_selection_2, sizeof(int64_t) * main_3_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_3_selection_2, -1, sizeof(int64_t) * main_3_selection_2_cpw_size);
int64_t* d_cycles_per_warp_main_3_semi_join_build_4;
auto main_3_semi_join_build_4_cpw_size = std::ceil((float)COUNT0/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_semi_join_build_4, sizeof(int64_t) * main_3_semi_join_build_4_cpw_size);
hipMemset(d_cycles_per_warp_main_3_semi_join_build_4, -1, sizeof(int64_t) * main_3_semi_join_build_4_cpw_size);
size_t COUNT4 = COUNT0;
// Insert hash table control;
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)COUNT0/(float)TILE_SIZE), TB>>>(COUNT0, d_HT_4.ref(cuco::insert), d_aggr0__tmp_attr0, d_cycles_per_warp_main_3_selection_2, d_cycles_per_warp_main_3_semi_join_build_4, d_KEY_0lineitem_u_1__l_orderkey);
int64_t* d_cycles_per_warp_main_7_join_build_6;
auto main_7_join_build_6_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_build_6, sizeof(int64_t) * main_7_join_build_6_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_build_6, -1, sizeof(int64_t) * main_7_join_build_6_cpw_size);
size_t COUNT6 = customer_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 1);
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)customer_size/(float)TILE_SIZE), TB>>>(d_BUF_6, d_BUF_IDX_6, d_HT_6.ref(cuco::insert), d_customer__c_custkey, customer_size, d_cycles_per_warp_main_7_join_build_6);
int64_t* cycles_per_warp_main_7_join_build_6 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_build_6_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_build_6, d_cycles_per_warp_main_7_join_build_6, sizeof(int64_t) * main_7_join_build_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_build_6 ";
for (auto i=0ull; i < main_7_join_build_6_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_build_6[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_5_semi_join_probe_4;
auto main_5_semi_join_probe_4_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_semi_join_probe_4, sizeof(int64_t) * main_5_semi_join_probe_4_cpw_size);
hipMemset(d_cycles_per_warp_main_5_semi_join_probe_4, -1, sizeof(int64_t) * main_5_semi_join_probe_4_cpw_size);
int64_t* d_cycles_per_warp_main_5_join_probe_6;
auto main_5_join_probe_6_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_join_probe_6, sizeof(int64_t) * main_5_join_probe_6_cpw_size);
hipMemset(d_cycles_per_warp_main_5_join_probe_6, -1, sizeof(int64_t) * main_5_join_probe_6_cpw_size);
int64_t* d_cycles_per_warp_main_5_join_build_8;
auto main_5_join_build_8_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_join_build_8, sizeof(int64_t) * main_5_join_build_8_cpw_size);
hipMemset(d_cycles_per_warp_main_5_join_build_8, -1, sizeof(int64_t) * main_5_join_build_8_cpw_size);
size_t COUNT8 = orders_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_8;
hipMalloc(&d_BUF_IDX_8, sizeof(uint64_t));
hipMemset(d_BUF_IDX_8, 0, sizeof(uint64_t));
uint64_t* d_BUF_8;
hipMalloc(&d_BUF_8, sizeof(uint64_t) * COUNT8 * 2);
auto d_HT_8 = cuco::static_map{ (int)COUNT8*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)orders_size/(float)TILE_SIZE), TB>>>(d_BUF_6, d_BUF_8, d_BUF_IDX_8, d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::insert), d_cycles_per_warp_main_5_join_build_8, d_cycles_per_warp_main_5_join_probe_6, d_cycles_per_warp_main_5_semi_join_probe_4, d_orders__o_custkey, d_orders__o_orderkey, orders_size);
int64_t* cycles_per_warp_main_5_semi_join_probe_4 = (int64_t*)malloc(sizeof(int64_t) * main_5_semi_join_probe_4_cpw_size);
hipMemcpy(cycles_per_warp_main_5_semi_join_probe_4, d_cycles_per_warp_main_5_semi_join_probe_4, sizeof(int64_t) * main_5_semi_join_probe_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_semi_join_probe_4 ";
for (auto i=0ull; i < main_5_semi_join_probe_4_cpw_size; i++) std::cout << cycles_per_warp_main_5_semi_join_probe_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_join_probe_6 = (int64_t*)malloc(sizeof(int64_t) * main_5_join_probe_6_cpw_size);
hipMemcpy(cycles_per_warp_main_5_join_probe_6, d_cycles_per_warp_main_5_join_probe_6, sizeof(int64_t) * main_5_join_probe_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_join_probe_6 ";
for (auto i=0ull; i < main_5_join_probe_6_cpw_size; i++) std::cout << cycles_per_warp_main_5_join_probe_6[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_join_build_8 = (int64_t*)malloc(sizeof(int64_t) * main_5_join_build_8_cpw_size);
hipMemcpy(cycles_per_warp_main_5_join_build_8, d_cycles_per_warp_main_5_join_build_8, sizeof(int64_t) * main_5_join_build_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_join_build_8 ";
for (auto i=0ull; i < main_5_join_build_8_cpw_size; i++) std::cout << cycles_per_warp_main_5_join_build_8[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_9_join_probe_8;
auto main_9_join_probe_8_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_9_join_probe_8, sizeof(int64_t) * main_9_join_probe_8_cpw_size);
hipMemset(d_cycles_per_warp_main_9_join_probe_8, -1, sizeof(int64_t) * main_9_join_probe_8_cpw_size);
int64_t* d_cycles_per_warp_main_9_aggregation_10;
auto main_9_aggregation_10_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_9_aggregation_10, sizeof(int64_t) * main_9_aggregation_10_cpw_size);
hipMemset(d_cycles_per_warp_main_9_aggregation_10, -1, sizeof(int64_t) * main_9_aggregation_10_cpw_size);
size_t COUNT10 = 6001215;
auto d_HT_10 = cuco::static_map{ (int)6001215*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_10;
hipMalloc(&d_SLOT_COUNT_10, sizeof(int));
hipMemset(d_SLOT_COUNT_10, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr1__tmp_attr1;
hipMalloc(&d_aggr1__tmp_attr1, sizeof(DBDecimalType) * COUNT10);
hipMemset(d_aggr1__tmp_attr1, 0, sizeof(DBDecimalType) * COUNT10);
DBI32Type* d_aggr__c_custkey;
hipMalloc(&d_aggr__c_custkey, sizeof(DBI32Type) * COUNT10);
hipMemset(d_aggr__c_custkey, 0, sizeof(DBI32Type) * COUNT10);
DBDecimalType* d_aggr__o_totalprice;
hipMalloc(&d_aggr__o_totalprice, sizeof(DBDecimalType) * COUNT10);
hipMemset(d_aggr__o_totalprice, 0, sizeof(DBDecimalType) * COUNT10);
DBDateType* d_aggr__o_orderdate;
hipMalloc(&d_aggr__o_orderdate, sizeof(DBDateType) * COUNT10);
hipMemset(d_aggr__o_orderdate, 0, sizeof(DBDateType) * COUNT10);
DBI32Type* d_KEY_10orders__o_orderkey;
hipMalloc(&d_KEY_10orders__o_orderkey, sizeof(DBI32Type) * COUNT10);
hipMemset(d_KEY_10orders__o_orderkey, 0, sizeof(DBI32Type) * COUNT10);
main_9<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TB>>>(d_BUF_8, d_HT_10.ref(cuco::insert_and_find), d_HT_8.ref(cuco::find), d_KEY_10orders__o_orderkey, d_SLOT_COUNT_10, d_aggr1__tmp_attr1, d_aggr__c_custkey, d_aggr__o_orderdate, d_aggr__o_totalprice, d_customer__c_custkey, d_cycles_per_warp_main_9_aggregation_10, d_cycles_per_warp_main_9_join_probe_8, d_lineitem__l_orderkey, d_lineitem__l_quantity, lineitem_size, d_orders__o_orderdate, d_orders__o_orderkey, d_orders__o_totalprice);
int64_t* cycles_per_warp_main_9_join_probe_8 = (int64_t*)malloc(sizeof(int64_t) * main_9_join_probe_8_cpw_size);
hipMemcpy(cycles_per_warp_main_9_join_probe_8, d_cycles_per_warp_main_9_join_probe_8, sizeof(int64_t) * main_9_join_probe_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_9_join_probe_8 ";
for (auto i=0ull; i < main_9_join_probe_8_cpw_size; i++) std::cout << cycles_per_warp_main_9_join_probe_8[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_9_aggregation_10 = (int64_t*)malloc(sizeof(int64_t) * main_9_aggregation_10_cpw_size);
hipMemcpy(cycles_per_warp_main_9_aggregation_10, d_cycles_per_warp_main_9_aggregation_10, sizeof(int64_t) * main_9_aggregation_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_9_aggregation_10 ";
for (auto i=0ull; i < main_9_aggregation_10_cpw_size; i++) std::cout << cycles_per_warp_main_9_aggregation_10[i] << " ";
std::cout << std::endl;
COUNT10 = d_HT_10.size();
int64_t* d_cycles_per_warp_main_12_materialize_11;
auto main_12_materialize_11_cpw_size = std::ceil((float)COUNT10/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_12_materialize_11, sizeof(int64_t) * main_12_materialize_11_cpw_size);
hipMemset(d_cycles_per_warp_main_12_materialize_11, -1, sizeof(int64_t) * main_12_materialize_11_cpw_size);
size_t COUNT11 = COUNT10;
//Materialize buffers
uint64_t* d_MAT_IDX11;
hipMalloc(&d_MAT_IDX11, sizeof(uint64_t));
hipMemset(d_MAT_IDX11, 0, sizeof(uint64_t));
auto MAT11aggr__c_custkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT11);
DBI32Type* d_MAT11aggr__c_custkey;
hipMalloc(&d_MAT11aggr__c_custkey, sizeof(DBI32Type) * COUNT11);
auto MAT11orders__o_orderkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT11);
DBI32Type* d_MAT11orders__o_orderkey;
hipMalloc(&d_MAT11orders__o_orderkey, sizeof(DBI32Type) * COUNT11);
auto MAT11aggr__o_orderdate = (DBDateType*)malloc(sizeof(DBDateType) * COUNT11);
DBDateType* d_MAT11aggr__o_orderdate;
hipMalloc(&d_MAT11aggr__o_orderdate, sizeof(DBDateType) * COUNT11);
auto MAT11aggr__o_totalprice = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT11);
DBDecimalType* d_MAT11aggr__o_totalprice;
hipMalloc(&d_MAT11aggr__o_totalprice, sizeof(DBDecimalType) * COUNT11);
auto MAT11aggr1__tmp_attr1 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT11);
DBDecimalType* d_MAT11aggr1__tmp_attr1;
hipMalloc(&d_MAT11aggr1__tmp_attr1, sizeof(DBDecimalType) * COUNT11);
main_12<<<std::ceil((float)COUNT10/(float)TILE_SIZE), TB>>>(COUNT10, d_MAT11aggr1__tmp_attr1, d_MAT11aggr__c_custkey, d_MAT11aggr__o_orderdate, d_MAT11aggr__o_totalprice, d_MAT11orders__o_orderkey, d_MAT_IDX11, d_aggr1__tmp_attr1, d_aggr__c_custkey, d_aggr__o_orderdate, d_aggr__o_totalprice, d_cycles_per_warp_main_12_materialize_11, d_KEY_10orders__o_orderkey);
uint64_t MATCOUNT_11 = 0;
hipMemcpy(&MATCOUNT_11, d_MAT_IDX11, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT11aggr__c_custkey, d_MAT11aggr__c_custkey, sizeof(DBI32Type) * COUNT11, hipMemcpyDeviceToHost);
hipMemcpy(MAT11orders__o_orderkey, d_MAT11orders__o_orderkey, sizeof(DBI32Type) * COUNT11, hipMemcpyDeviceToHost);
hipMemcpy(MAT11aggr__o_orderdate, d_MAT11aggr__o_orderdate, sizeof(DBDateType) * COUNT11, hipMemcpyDeviceToHost);
hipMemcpy(MAT11aggr__o_totalprice, d_MAT11aggr__o_totalprice, sizeof(DBDecimalType) * COUNT11, hipMemcpyDeviceToHost);
hipMemcpy(MAT11aggr1__tmp_attr1, d_MAT11aggr1__tmp_attr1, sizeof(DBDecimalType) * COUNT11, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_12_materialize_11 = (int64_t*)malloc(sizeof(int64_t) * main_12_materialize_11_cpw_size);
hipMemcpy(cycles_per_warp_main_12_materialize_11, d_cycles_per_warp_main_12_materialize_11, sizeof(int64_t) * main_12_materialize_11_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_12_materialize_11 ";
for (auto i=0ull; i < main_12_materialize_11_cpw_size; i++) std::cout << cycles_per_warp_main_12_materialize_11[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_KEY_0lineitem_u_1__l_orderkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_BUF_8);
hipFree(d_BUF_IDX_8);
hipFree(d_KEY_10orders__o_orderkey);
hipFree(d_aggr1__tmp_attr1);
hipFree(d_aggr__c_custkey);
hipFree(d_aggr__o_orderdate);
hipFree(d_aggr__o_totalprice);
hipFree(d_MAT11aggr1__tmp_attr1);
hipFree(d_MAT11aggr__c_custkey);
hipFree(d_MAT11aggr__o_orderdate);
hipFree(d_MAT11aggr__o_totalprice);
hipFree(d_MAT11orders__o_orderkey);
hipFree(d_MAT_IDX11);
free(MAT11aggr1__tmp_attr1);
free(MAT11aggr__c_custkey);
free(MAT11aggr__o_orderdate);
free(MAT11aggr__o_totalprice);
free(MAT11orders__o_orderkey);
}