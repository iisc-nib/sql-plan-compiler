#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_1(uint64_t* COUNT0, size_t customer_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT0, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0[ITEM], buf_idx_0});
BUF_0[(buf_idx_0) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_3(uint64_t* BUF_0, HASHTABLE_PROBE HT_0, HASHTABLE_INSERT HT_2, DBI32Type* customer__c_custkey, DBI32Type* orders__o_custkey, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_custkey[ITEM] = orders__o_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_orders__o_custkey[ITEM];
}
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_custkey[ITEM] = customer__c_custkey[BUF_0[slot_second0[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_customer__c_custkey[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_2.insert(cuco::pair{KEY_2[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_3(uint64_t* BUF_0, HASHTABLE_PROBE HT_0, HASHTABLE_FIND HT_2, DBI32Type* KEY_2customer__c_custkey, DBI64Type* aggr0__tmp_attr0, DBI32Type* customer__c_custkey, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_custkey[ITEM] = orders__o_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_orders__o_custkey[ITEM];
}
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_custkey[ITEM] = customer__c_custkey[BUF_0[slot_second0[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_customer__c_custkey[ITEM];
}
//Aggregate in hashtable
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_2 = HT_2.find(KEY_2[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr0[buf_idx_2], 1);
KEY_2customer__c_custkey[buf_idx_2] = reg_customer__c_custkey[ITEM];
}
}
template<typename HASHTABLE_INSERT>
__global__ void count_5(size_t COUNT2, HASHTABLE_INSERT HT_4, DBI64Type* aggr0__tmp_attr0) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI64Type reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= (DBI32Type)reg_aggr0__tmp_attr0[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_4.insert(cuco::pair{KEY_4[ITEM], 1});
}
}
template<typename HASHTABLE_FIND>
__global__ void main_5(size_t COUNT2, HASHTABLE_FIND HT_4, DBI64Type* KEY_4aggr0__tmp_attr0, DBI64Type* aggr0__tmp_attr0, DBI64Type* aggr1__tmp_attr1) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI64Type reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= (DBI32Type)reg_aggr0__tmp_attr0[ITEM];
}
//Aggregate in hashtable
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_4 = HT_4.find(KEY_4[ITEM])->second;
aggregate_sum(&aggr1__tmp_attr1[buf_idx_4], 1);
KEY_4aggr0__tmp_attr0[buf_idx_4] = reg_aggr0__tmp_attr0[ITEM];
}
}
__global__ void count_7(size_t COUNT4, uint64_t* COUNT6) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT6, 1);
}
}
__global__ void main_7(size_t COUNT4, DBI64Type* MAT6aggr0__tmp_attr0, DBI64Type* MAT6aggr1__tmp_attr1, uint64_t* MAT_IDX6, DBI64Type* aggr0__tmp_attr0, DBI64Type* aggr1__tmp_attr1) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize buffers
DBI64Type reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
DBI64Type reg_aggr1__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr1__tmp_attr1[ITEM] = aggr1__tmp_attr1[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx6 = atomicAdd((int*)MAT_IDX6, 1);
MAT6aggr0__tmp_attr0[mat_idx6] = reg_aggr0__tmp_attr0[ITEM];
MAT6aggr1__tmp_attr1[mat_idx6] = reg_aggr1__tmp_attr1[ITEM];
}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map) {
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT0, customer_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_customer__c_custkey, customer_size);
//Create aggregation hash table
auto d_HT_2 = cuco::static_map{ (int)1500000*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_3<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::insert), d_customer__c_custkey, d_orders__o_custkey, orders_size);
size_t COUNT2 = d_HT_2.size();
thrust::device_vector<int64_t> keys_2(COUNT2), vals_2(COUNT2);
d_HT_2.retrieve_all(keys_2.begin(), vals_2.begin());
d_HT_2.clear();
int64_t* raw_keys2 = thrust::raw_pointer_cast(keys_2.data());
insertKeys<<<std::ceil((float)COUNT2/128.), 128>>>(raw_keys2, d_HT_2.ref(cuco::insert), COUNT2);
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT2);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT2);
DBI32Type* d_KEY_2customer__c_custkey;
hipMalloc(&d_KEY_2customer__c_custkey, sizeof(DBI32Type) * COUNT2);
hipMemset(d_KEY_2customer__c_custkey, 0, sizeof(DBI32Type) * COUNT2);
main_3<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::find), d_KEY_2customer__c_custkey, d_aggr0__tmp_attr0, d_customer__c_custkey, d_orders__o_custkey, d_orders__o_orderkey, orders_size);
//Create aggregation hash table
auto d_HT_4 = cuco::static_map{ (int)1500000*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5<<<std::ceil((float)COUNT2/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT2, d_HT_4.ref(cuco::insert), d_aggr0__tmp_attr0);
size_t COUNT4 = d_HT_4.size();
thrust::device_vector<int64_t> keys_4(COUNT4), vals_4(COUNT4);
d_HT_4.retrieve_all(keys_4.begin(), vals_4.begin());
d_HT_4.clear();
int64_t* raw_keys4 = thrust::raw_pointer_cast(keys_4.data());
insertKeys<<<std::ceil((float)COUNT4/128.), 128>>>(raw_keys4, d_HT_4.ref(cuco::insert), COUNT4);
//Aggregate in hashtable
DBI64Type* d_aggr1__tmp_attr1;
hipMalloc(&d_aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT4);
hipMemset(d_aggr1__tmp_attr1, 0, sizeof(DBI64Type) * COUNT4);
DBI64Type* d_KEY_4aggr0__tmp_attr0;
hipMalloc(&d_KEY_4aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT4);
hipMemset(d_KEY_4aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT4);
main_5<<<std::ceil((float)COUNT2/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT2, d_HT_4.ref(cuco::find), d_KEY_4aggr0__tmp_attr0, d_aggr0__tmp_attr0, d_aggr1__tmp_attr1);
//Materialize count
uint64_t* d_COUNT6;
hipMalloc(&d_COUNT6, sizeof(uint64_t));
hipMemset(d_COUNT6, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)COUNT4/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT4, d_COUNT6);
uint64_t COUNT6;
hipMemcpy(&COUNT6, d_COUNT6, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX6;
hipMalloc(&d_MAT_IDX6, sizeof(uint64_t));
hipMemset(d_MAT_IDX6, 0, sizeof(uint64_t));
auto MAT6aggr0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT6);
DBI64Type* d_MAT6aggr0__tmp_attr0;
hipMalloc(&d_MAT6aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT6);
auto MAT6aggr1__tmp_attr1 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT6);
DBI64Type* d_MAT6aggr1__tmp_attr1;
hipMalloc(&d_MAT6aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT6);
main_7<<<std::ceil((float)COUNT4/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT4, d_MAT6aggr0__tmp_attr0, d_MAT6aggr1__tmp_attr1, d_MAT_IDX6, d_KEY_4aggr0__tmp_attr0, d_aggr1__tmp_attr1);
hipMemcpy(MAT6aggr0__tmp_attr0, d_MAT6aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT6, hipMemcpyDeviceToHost);
hipMemcpy(MAT6aggr1__tmp_attr1, d_MAT6aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT6, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT6; i++) { std::cout << "" << MAT6aggr0__tmp_attr0[i];
std::cout << "," << MAT6aggr1__tmp_attr1[i];
std::cout << std::endl; }
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_COUNT0);
hipFree(d_KEY_2customer__c_custkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_KEY_4aggr0__tmp_attr0);
hipFree(d_aggr1__tmp_attr1);
hipFree(d_COUNT6);
hipFree(d_MAT6aggr0__tmp_attr0);
hipFree(d_MAT6aggr1__tmp_attr1);
hipFree(d_MAT_IDX6);
free(MAT6aggr0__tmp_attr0);
free(MAT6aggr1__tmp_attr1);
}