#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_59c0b9dd83d0(uint64_t* COUNT59c0b9dea900, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8674, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::lt))) return;
//Materialize count
atomicAdd((int*)COUNT59c0b9dea900, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_59c0b9dd83d0(uint64_t* BUF_59c0b9dea900, uint64_t* BUF_IDX_59c0b9dea900, HASHTABLE_INSERT HT_59c0b9dea900, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8674, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::lt))) return;
uint64_t KEY_59c0b9dea900 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_59c0b9dea900 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_59c0b9dea900 = atomicAdd((int*)BUF_IDX_59c0b9dea900, 1);
HT_59c0b9dea900.insert(cuco::pair{KEY_59c0b9dea900, buf_idx_59c0b9dea900});
BUF_59c0b9dea900[buf_idx_59c0b9dea900 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_59c0b9df6db0(uint64_t* BUF_59c0b9dea900, uint64_t* COUNT59c0b9dea7b0, HASHTABLE_PROBE HT_59c0b9dea900, DBI32Type* lineitem__l_orderkey, DBCharType* lineitem__l_returnflag, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];
if (!(evaluatePredicate(reg_lineitem__l_returnflag, 'R', Predicate::eq))) return;
uint64_t KEY_59c0b9dea900 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_59c0b9dea900 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_59c0b9dea900.for_each(KEY_59c0b9dea900, [&] __device__ (auto const SLOT_59c0b9dea900) {

auto const [slot_first59c0b9dea900, slot_second59c0b9dea900] = SLOT_59c0b9dea900;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT59c0b9dea7b0, 1);
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_59c0b9df6db0(uint64_t* BUF_59c0b9dea7b0, uint64_t* BUF_59c0b9dea900, uint64_t* BUF_IDX_59c0b9dea7b0, HASHTABLE_INSERT HT_59c0b9dea7b0, HASHTABLE_PROBE HT_59c0b9dea900, DBI32Type* lineitem__l_orderkey, DBCharType* lineitem__l_returnflag, size_t lineitem_size, DBI32Type* orders__o_custkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];
if (!(evaluatePredicate(reg_lineitem__l_returnflag, 'R', Predicate::eq))) return;
uint64_t KEY_59c0b9dea900 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_59c0b9dea900 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_59c0b9dea900.for_each(KEY_59c0b9dea900, [&] __device__ (auto const SLOT_59c0b9dea900) {
auto const [slot_first59c0b9dea900, slot_second59c0b9dea900] = SLOT_59c0b9dea900;
if (!(true)) return;
uint64_t KEY_59c0b9dea7b0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[BUF_59c0b9dea900[slot_second59c0b9dea900 * 1 + 0]];

KEY_59c0b9dea7b0 |= reg_orders__o_custkey;
// Insert hash table kernel;
auto buf_idx_59c0b9dea7b0 = atomicAdd((int*)BUF_IDX_59c0b9dea7b0, 1);
HT_59c0b9dea7b0.insert(cuco::pair{KEY_59c0b9dea7b0, buf_idx_59c0b9dea7b0});
BUF_59c0b9dea7b0[buf_idx_59c0b9dea7b0 * 2 + 0] = BUF_59c0b9dea900[slot_second59c0b9dea900 * 1 + 0];
BUF_59c0b9dea7b0[buf_idx_59c0b9dea7b0 * 2 + 1] = tid;
});
}
__global__ void count_59c0b9e01e90(uint64_t* COUNT59c0b9dec1b0, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
//Materialize count
atomicAdd((int*)COUNT59c0b9dec1b0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_59c0b9e01e90(uint64_t* BUF_59c0b9dec1b0, uint64_t* BUF_IDX_59c0b9dec1b0, HASHTABLE_INSERT HT_59c0b9dec1b0, DBI32Type* nation__n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_59c0b9dec1b0 = 0;
auto reg_nation__n_nationkey = nation__n_nationkey[tid];

KEY_59c0b9dec1b0 |= reg_nation__n_nationkey;
// Insert hash table kernel;
auto buf_idx_59c0b9dec1b0 = atomicAdd((int*)BUF_IDX_59c0b9dec1b0, 1);
HT_59c0b9dec1b0.insert(cuco::pair{KEY_59c0b9dec1b0, buf_idx_59c0b9dec1b0});
BUF_59c0b9dec1b0[buf_idx_59c0b9dec1b0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_59c0b9dd8040(uint64_t* BUF_59c0b9dea7b0, uint64_t* BUF_59c0b9dec1b0, HASHTABLE_INSERT HT_59c0b9da7b00, HASHTABLE_PROBE HT_59c0b9dea7b0, HASHTABLE_PROBE HT_59c0b9dec1b0, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_59c0b9dea7b0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_59c0b9dea7b0 |= reg_customer__c_custkey;
//Probe Hash table
HT_59c0b9dea7b0.for_each(KEY_59c0b9dea7b0, [&] __device__ (auto const SLOT_59c0b9dea7b0) {

auto const [slot_first59c0b9dea7b0, slot_second59c0b9dea7b0] = SLOT_59c0b9dea7b0;
if (!(true)) return;
uint64_t KEY_59c0b9dec1b0 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_59c0b9dec1b0 |= reg_customer__c_nationkey;
//Probe Hash table
HT_59c0b9dec1b0.for_each(KEY_59c0b9dec1b0, [&] __device__ (auto const SLOT_59c0b9dec1b0) {

auto const [slot_first59c0b9dec1b0, slot_second59c0b9dec1b0] = SLOT_59c0b9dec1b0;
if (!(true)) return;
uint64_t KEY_59c0b9da7b00 = 0;

KEY_59c0b9da7b00 |= reg_customer__c_custkey;
//Create aggregation hash table
HT_59c0b9da7b00.insert(cuco::pair{KEY_59c0b9da7b00, 1});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_59c0b9dd8040(uint64_t* BUF_59c0b9dea7b0, uint64_t* BUF_59c0b9dec1b0, HASHTABLE_FIND HT_59c0b9da7b00, HASHTABLE_PROBE HT_59c0b9dea7b0, HASHTABLE_PROBE HT_59c0b9dec1b0, DBI32Type* KEY_59c0b9da7b00customer__c_custkey, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr__c_acctbal, DBDecimalType* customer__c_acctbal, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_59c0b9dea7b0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_59c0b9dea7b0 |= reg_customer__c_custkey;
//Probe Hash table
HT_59c0b9dea7b0.for_each(KEY_59c0b9dea7b0, [&] __device__ (auto const SLOT_59c0b9dea7b0) {
auto const [slot_first59c0b9dea7b0, slot_second59c0b9dea7b0] = SLOT_59c0b9dea7b0;
if (!(true)) return;
uint64_t KEY_59c0b9dec1b0 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_59c0b9dec1b0 |= reg_customer__c_nationkey;
//Probe Hash table
HT_59c0b9dec1b0.for_each(KEY_59c0b9dec1b0, [&] __device__ (auto const SLOT_59c0b9dec1b0) {
auto const [slot_first59c0b9dec1b0, slot_second59c0b9dec1b0] = SLOT_59c0b9dec1b0;
if (!(true)) return;
uint64_t KEY_59c0b9da7b00 = 0;

KEY_59c0b9da7b00 |= reg_customer__c_custkey;
//Aggregate in hashtable
auto buf_idx_59c0b9da7b00 = HT_59c0b9da7b00.find(KEY_59c0b9da7b00)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[BUF_59c0b9dea7b0[slot_second59c0b9dea7b0 * 2 + 1]];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[BUF_59c0b9dea7b0[slot_second59c0b9dea7b0 * 2 + 1]];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_59c0b9da7b00], reg_map0__tmp_attr1);
auto reg_customer__c_acctbal = customer__c_acctbal[tid];
aggregate_any(&aggr__c_acctbal[buf_idx_59c0b9da7b00], reg_customer__c_acctbal);
KEY_59c0b9da7b00customer__c_custkey[buf_idx_59c0b9da7b00] = reg_customer__c_custkey;
});
});
}
__global__ void count_59c0b9e06620(uint64_t* COUNT59c0b9d85c70, size_t COUNT59c0b9da7b00) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT59c0b9da7b00) return;
//Materialize count
atomicAdd((int*)COUNT59c0b9d85c70, 1);
}
__global__ void main_59c0b9e06620(size_t COUNT59c0b9da7b00, DBDecimalType* MAT59c0b9d85c70aggr0__tmp_attr0, DBDecimalType* MAT59c0b9d85c70aggr__c_acctbal, DBI32Type* MAT59c0b9d85c70customer__c_custkey, uint64_t* MAT_IDX59c0b9d85c70, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr__c_acctbal, DBI32Type* customer__c_custkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT59c0b9da7b00) return;
//Materialize buffers
auto mat_idx59c0b9d85c70 = atomicAdd((int*)MAT_IDX59c0b9d85c70, 1);
auto reg_customer__c_custkey = customer__c_custkey[tid];
MAT59c0b9d85c70customer__c_custkey[mat_idx59c0b9d85c70] = reg_customer__c_custkey;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT59c0b9d85c70aggr0__tmp_attr0[mat_idx59c0b9d85c70] = reg_aggr0__tmp_attr0;
auto reg_aggr__c_acctbal = aggr__c_acctbal[tid];
MAT59c0b9d85c70aggr__c_acctbal[mat_idx59c0b9d85c70] = reg_aggr__c_acctbal;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map) {
//Materialize count
uint64_t* d_COUNT59c0b9dea900;
hipMalloc(&d_COUNT59c0b9dea900, sizeof(uint64_t));
hipMemset(d_COUNT59c0b9dea900, 0, sizeof(uint64_t));
count_59c0b9dd83d0<<<std::ceil((float)orders_size/32.), 32>>>(d_COUNT59c0b9dea900, d_orders__o_orderdate, orders_size);
uint64_t COUNT59c0b9dea900;
hipMemcpy(&COUNT59c0b9dea900, d_COUNT59c0b9dea900, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_59c0b9dea900;
hipMalloc(&d_BUF_IDX_59c0b9dea900, sizeof(uint64_t));
hipMemset(d_BUF_IDX_59c0b9dea900, 0, sizeof(uint64_t));
uint64_t* d_BUF_59c0b9dea900;
hipMalloc(&d_BUF_59c0b9dea900, sizeof(uint64_t) * COUNT59c0b9dea900 * 1);
auto d_HT_59c0b9dea900 = cuco::experimental::static_multimap{ (int)COUNT59c0b9dea900*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_59c0b9dd83d0<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_59c0b9dea900, d_BUF_IDX_59c0b9dea900, d_HT_59c0b9dea900.ref(cuco::insert), d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
//Materialize count
uint64_t* d_COUNT59c0b9dea7b0;
hipMalloc(&d_COUNT59c0b9dea7b0, sizeof(uint64_t));
hipMemset(d_COUNT59c0b9dea7b0, 0, sizeof(uint64_t));
count_59c0b9df6db0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_59c0b9dea900, d_COUNT59c0b9dea7b0, d_HT_59c0b9dea900.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_returnflag, lineitem_size);
uint64_t COUNT59c0b9dea7b0;
hipMemcpy(&COUNT59c0b9dea7b0, d_COUNT59c0b9dea7b0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_59c0b9dea7b0;
hipMalloc(&d_BUF_IDX_59c0b9dea7b0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_59c0b9dea7b0, 0, sizeof(uint64_t));
uint64_t* d_BUF_59c0b9dea7b0;
hipMalloc(&d_BUF_59c0b9dea7b0, sizeof(uint64_t) * COUNT59c0b9dea7b0 * 2);
auto d_HT_59c0b9dea7b0 = cuco::experimental::static_multimap{ (int)COUNT59c0b9dea7b0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_59c0b9df6db0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_59c0b9dea7b0, d_BUF_59c0b9dea900, d_BUF_IDX_59c0b9dea7b0, d_HT_59c0b9dea7b0.ref(cuco::insert), d_HT_59c0b9dea900.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_returnflag, lineitem_size, d_orders__o_custkey);
//Materialize count
uint64_t* d_COUNT59c0b9dec1b0;
hipMalloc(&d_COUNT59c0b9dec1b0, sizeof(uint64_t));
hipMemset(d_COUNT59c0b9dec1b0, 0, sizeof(uint64_t));
count_59c0b9e01e90<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT59c0b9dec1b0, nation_size);
uint64_t COUNT59c0b9dec1b0;
hipMemcpy(&COUNT59c0b9dec1b0, d_COUNT59c0b9dec1b0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_59c0b9dec1b0;
hipMalloc(&d_BUF_IDX_59c0b9dec1b0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_59c0b9dec1b0, 0, sizeof(uint64_t));
uint64_t* d_BUF_59c0b9dec1b0;
hipMalloc(&d_BUF_59c0b9dec1b0, sizeof(uint64_t) * COUNT59c0b9dec1b0 * 1);
auto d_HT_59c0b9dec1b0 = cuco::experimental::static_multimap{ (int)COUNT59c0b9dec1b0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_59c0b9e01e90<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_59c0b9dec1b0, d_BUF_IDX_59c0b9dec1b0, d_HT_59c0b9dec1b0.ref(cuco::insert), d_nation__n_nationkey, nation_size);
//Create aggregation hash table
auto d_HT_59c0b9da7b00 = cuco::static_map{ (int)45145*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_59c0b9dd8040<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_59c0b9dea7b0, d_BUF_59c0b9dec1b0, d_HT_59c0b9da7b00.ref(cuco::insert), d_HT_59c0b9dea7b0.ref(cuco::for_each), d_HT_59c0b9dec1b0.ref(cuco::for_each), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
size_t COUNT59c0b9da7b00 = d_HT_59c0b9da7b00.size();
thrust::device_vector<int64_t> keys_59c0b9da7b00(COUNT59c0b9da7b00), vals_59c0b9da7b00(COUNT59c0b9da7b00);
d_HT_59c0b9da7b00.retrieve_all(keys_59c0b9da7b00.begin(), vals_59c0b9da7b00.begin());
d_HT_59c0b9da7b00.clear();
int64_t* raw_keys59c0b9da7b00 = thrust::raw_pointer_cast(keys_59c0b9da7b00.data());
insertKeys<<<std::ceil((float)COUNT59c0b9da7b00/32.), 32>>>(raw_keys59c0b9da7b00, d_HT_59c0b9da7b00.ref(cuco::insert), COUNT59c0b9da7b00);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT59c0b9da7b00);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT59c0b9da7b00);
DBDecimalType* d_aggr__c_acctbal;
hipMalloc(&d_aggr__c_acctbal, sizeof(DBDecimalType) * COUNT59c0b9da7b00);
hipMemset(d_aggr__c_acctbal, 0, sizeof(DBDecimalType) * COUNT59c0b9da7b00);
DBI32Type* d_KEY_59c0b9da7b00customer__c_custkey;
hipMalloc(&d_KEY_59c0b9da7b00customer__c_custkey, sizeof(DBI32Type) * COUNT59c0b9da7b00);
hipMemset(d_KEY_59c0b9da7b00customer__c_custkey, 0, sizeof(DBI32Type) * COUNT59c0b9da7b00);
main_59c0b9dd8040<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_59c0b9dea7b0, d_BUF_59c0b9dec1b0, d_HT_59c0b9da7b00.ref(cuco::find), d_HT_59c0b9dea7b0.ref(cuco::for_each), d_HT_59c0b9dec1b0.ref(cuco::for_each), d_KEY_59c0b9da7b00customer__c_custkey, d_aggr0__tmp_attr0, d_aggr__c_acctbal, d_customer__c_acctbal, d_customer__c_custkey, d_customer__c_nationkey, customer_size, d_lineitem__l_discount, d_lineitem__l_extendedprice);
//Materialize count
uint64_t* d_COUNT59c0b9d85c70;
hipMalloc(&d_COUNT59c0b9d85c70, sizeof(uint64_t));
hipMemset(d_COUNT59c0b9d85c70, 0, sizeof(uint64_t));
count_59c0b9e06620<<<std::ceil((float)COUNT59c0b9da7b00/32.), 32>>>(d_COUNT59c0b9d85c70, COUNT59c0b9da7b00);
uint64_t COUNT59c0b9d85c70;
hipMemcpy(&COUNT59c0b9d85c70, d_COUNT59c0b9d85c70, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX59c0b9d85c70;
hipMalloc(&d_MAT_IDX59c0b9d85c70, sizeof(uint64_t));
hipMemset(d_MAT_IDX59c0b9d85c70, 0, sizeof(uint64_t));
auto MAT59c0b9d85c70customer__c_custkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT59c0b9d85c70);
DBI32Type* d_MAT59c0b9d85c70customer__c_custkey;
hipMalloc(&d_MAT59c0b9d85c70customer__c_custkey, sizeof(DBI32Type) * COUNT59c0b9d85c70);
auto MAT59c0b9d85c70aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT59c0b9d85c70);
DBDecimalType* d_MAT59c0b9d85c70aggr0__tmp_attr0;
hipMalloc(&d_MAT59c0b9d85c70aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT59c0b9d85c70);
auto MAT59c0b9d85c70aggr__c_acctbal = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT59c0b9d85c70);
DBDecimalType* d_MAT59c0b9d85c70aggr__c_acctbal;
hipMalloc(&d_MAT59c0b9d85c70aggr__c_acctbal, sizeof(DBDecimalType) * COUNT59c0b9d85c70);
main_59c0b9e06620<<<std::ceil((float)COUNT59c0b9da7b00/32.), 32>>>(COUNT59c0b9da7b00, d_MAT59c0b9d85c70aggr0__tmp_attr0, d_MAT59c0b9d85c70aggr__c_acctbal, d_MAT59c0b9d85c70customer__c_custkey, d_MAT_IDX59c0b9d85c70, d_aggr0__tmp_attr0, d_aggr__c_acctbal, d_KEY_59c0b9da7b00customer__c_custkey);
hipMemcpy(MAT59c0b9d85c70customer__c_custkey, d_MAT59c0b9d85c70customer__c_custkey, sizeof(DBI32Type) * COUNT59c0b9d85c70, hipMemcpyDeviceToHost);
hipMemcpy(MAT59c0b9d85c70aggr0__tmp_attr0, d_MAT59c0b9d85c70aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT59c0b9d85c70, hipMemcpyDeviceToHost);
hipMemcpy(MAT59c0b9d85c70aggr__c_acctbal, d_MAT59c0b9d85c70aggr__c_acctbal, sizeof(DBDecimalType) * COUNT59c0b9d85c70, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT59c0b9d85c70; i++) { std::cout << MAT59c0b9d85c70customer__c_custkey[i] << "\t";
std::cout << MAT59c0b9d85c70aggr0__tmp_attr0[i] << "\t";
std::cout << MAT59c0b9d85c70aggr__c_acctbal[i] << "\t";
std::cout << std::endl; }
std::cout << COUNT59c0b9d85c70 << std::endl;
hipFree(d_BUF_59c0b9dea900);
hipFree(d_BUF_IDX_59c0b9dea900);
hipFree(d_COUNT59c0b9dea900);
hipFree(d_BUF_59c0b9dea7b0);
hipFree(d_BUF_IDX_59c0b9dea7b0);
hipFree(d_COUNT59c0b9dea7b0);
hipFree(d_BUF_59c0b9dec1b0);
hipFree(d_BUF_IDX_59c0b9dec1b0);
hipFree(d_COUNT59c0b9dec1b0);
hipFree(d_KEY_59c0b9da7b00customer__c_custkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr__c_acctbal);
hipFree(d_COUNT59c0b9d85c70);
hipFree(d_MAT59c0b9d85c70aggr0__tmp_attr0);
hipFree(d_MAT59c0b9d85c70aggr__c_acctbal);
hipFree(d_MAT59c0b9d85c70customer__c_custkey);
hipFree(d_MAT_IDX59c0b9d85c70);
free(MAT59c0b9d85c70aggr0__tmp_attr0);
free(MAT59c0b9d85c70aggr__c_acctbal);
free(MAT59c0b9d85c70customer__c_custkey);
}