#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_1(uint64_t* COUNT0, DBStringType* region__r_name, size_t region_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_region__r_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
reg_region__r_name[ITEM] = region__r_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_region__r_name[ITEM], "ASIA", Predicate::eq);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT0, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBStringType* region__r_name, DBI32Type* region__r_regionkey, size_t region_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_region__r_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
reg_region__r_name[ITEM] = region__r_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_region__r_name[ITEM], "ASIA", Predicate::eq);
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_region__r_regionkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
reg_region__r_regionkey[ITEM] = region__r_regionkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_region__r_regionkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0[ITEM], buf_idx_0});
BUF_0[(buf_idx_0) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE>
__global__ void count_3(uint64_t* BUF_0, uint64_t* COUNT2, HASHTABLE_PROBE HT_0, DBI32Type* nation__n_regionkey, size_t nation_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_nation__n_regionkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_nation__n_regionkey[ITEM] = nation__n_regionkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_nation__n_regionkey[ITEM];
}
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT2, 1);
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_0, uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_PROBE HT_0, HASHTABLE_INSERT HT_2, DBI32Type* nation__n_nationkey, DBI32Type* nation__n_regionkey, size_t nation_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_nation__n_regionkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_nation__n_regionkey[ITEM] = nation__n_regionkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_nation__n_regionkey[ITEM];
}
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_nation__n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_nation__n_nationkey[ITEM] = nation__n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_nation__n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_2 = atomicAdd((int*)BUF_IDX_2, 1);
HT_2.insert(cuco::pair{KEY_2[ITEM], buf_idx_2});
BUF_2[(buf_idx_2) * 2 + 0] = BUF_0[slot_second0[ITEM] * 1 + 0];
BUF_2[(buf_idx_2) * 2 + 1] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE>
__global__ void count_5(uint64_t* BUF_2, uint64_t* COUNT4, HASHTABLE_PROBE HT_2, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_customer__c_nationkey[ITEM];
}
int64_t slot_second2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_2 = HT_2.find(KEY_2[ITEM]);
if (SLOT_2 == HT_2.end()) {selection_flags[ITEM] = 0; continue;}
slot_second2[ITEM] = SLOT_2->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT4, 1);
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_2, uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_PROBE HT_2, HASHTABLE_INSERT HT_4, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_customer__c_nationkey[ITEM];
}
int64_t slot_second2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_2 = HT_2.find(KEY_2[ITEM]);
if (SLOT_2 == HT_2.end()) {selection_flags[ITEM] = 0; continue;}
slot_second2[ITEM] = SLOT_2->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_4 = atomicAdd((int*)BUF_IDX_4, 1);
HT_4.insert(cuco::pair{KEY_4[ITEM], buf_idx_4});
BUF_4[(buf_idx_4) * 3 + 0] = ITEM*TB + tid;
BUF_4[(buf_idx_4) * 3 + 1] = BUF_2[slot_second2[ITEM] * 2 + 0];
BUF_4[(buf_idx_4) * 3 + 2] = BUF_2[slot_second2[ITEM] * 2 + 1];
}
}
template<typename HASHTABLE_PROBE>
__global__ void count_7(uint64_t* BUF_4, uint64_t* COUNT6, HASHTABLE_PROBE HT_4, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_orders__o_orderdate[ITEM], 8766, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate[ITEM], 9131, Predicate::lt);
}
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_custkey[ITEM] = orders__o_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_orders__o_custkey[ITEM];
}
int64_t slot_second4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0; continue;}
slot_second4[ITEM] = SLOT_4->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT6, 1);
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_7(uint64_t* BUF_4, uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_PROBE HT_4, HASHTABLE_INSERT HT_6, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_orders__o_orderdate[ITEM], 8766, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate[ITEM], 9131, Predicate::lt);
}
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_custkey[ITEM] = orders__o_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_orders__o_custkey[ITEM];
}
int64_t slot_second4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0; continue;}
slot_second4[ITEM] = SLOT_4->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_orders__o_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_6 = atomicAdd((int*)BUF_IDX_6, 1);
HT_6.insert(cuco::pair{KEY_6[ITEM], buf_idx_6});
BUF_6[(buf_idx_6) * 4 + 0] = BUF_4[slot_second4[ITEM] * 3 + 0];
BUF_6[(buf_idx_6) * 4 + 1] = ITEM*TB + tid;
BUF_6[(buf_idx_6) * 4 + 2] = BUF_4[slot_second4[ITEM] * 3 + 1];
BUF_6[(buf_idx_6) * 4 + 3] = BUF_4[slot_second4[ITEM] * 3 + 2];
}
}
__global__ void count_9(uint64_t* COUNT8, size_t supplier_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT8, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_9(uint64_t* BUF_8, uint64_t* BUF_IDX_8, HASHTABLE_INSERT HT_8, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
DBI32Type reg_supplier__s_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_nationkey[ITEM] = supplier__s_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_supplier__s_suppkey[ITEM];
KEY_8[ITEM] <<= 32;
KEY_8[ITEM] |= reg_supplier__s_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_8 = atomicAdd((int*)BUF_IDX_8, 1);
HT_8.insert(cuco::pair{KEY_8[ITEM], buf_idx_8});
BUF_8[(buf_idx_8) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_11(uint64_t* BUF_6, uint64_t* BUF_8, HASHTABLE_INSERT HT_10, HASHTABLE_PROBE HT_6, HASHTABLE_PROBE HT_8, DBI32Type* customer__c_nationkey, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_suppkey[ITEM] = lineitem__l_suppkey[ITEM*TB + tid];
}
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[BUF_6[slot_second6[ITEM] * 4 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_lineitem__l_suppkey[ITEM];
KEY_8[ITEM] <<= 32;
KEY_8[ITEM] |= reg_customer__c_nationkey[ITEM];
}
int64_t slot_second8[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_8 = HT_8.find(KEY_8[ITEM]);
if (SLOT_8 == HT_8.end()) {selection_flags[ITEM] = 0; continue;}
slot_second8[ITEM] = SLOT_8->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI16Type reg_nation__n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_nation__n_name_encoded[ITEM] = nation__n_name_encoded[BUF_6[slot_second6[ITEM] * 4 + 3]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_nation__n_name_encoded[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_10.insert(cuco::pair{KEY_10[ITEM], 1});
}
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_11(uint64_t* BUF_6, uint64_t* BUF_8, HASHTABLE_FIND HT_10, HASHTABLE_PROBE HT_6, HASHTABLE_PROBE HT_8, DBI16Type* KEY_10nation__n_name_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* customer__c_nationkey, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_suppkey[ITEM] = lineitem__l_suppkey[ITEM*TB + tid];
}
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[BUF_6[slot_second6[ITEM] * 4 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_lineitem__l_suppkey[ITEM];
KEY_8[ITEM] <<= 32;
KEY_8[ITEM] |= reg_customer__c_nationkey[ITEM];
}
int64_t slot_second8[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_8 = HT_8.find(KEY_8[ITEM]);
if (SLOT_8 == HT_8.end()) {selection_flags[ITEM] = 0; continue;}
slot_second8[ITEM] = SLOT_8->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI16Type reg_nation__n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_nation__n_name_encoded[ITEM] = nation__n_name_encoded[BUF_6[slot_second6[ITEM] * 4 + 3]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_nation__n_name_encoded[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineitem__l_extendedprice[ITEM]) * ((1) - (reg_lineitem__l_discount[ITEM]));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_10 = HT_10.find(KEY_10[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr0[buf_idx_10], reg_map0__tmp_attr1[ITEM]);
KEY_10nation__n_name_encoded[buf_idx_10] = reg_nation__n_name_encoded[ITEM];
}
}
__global__ void count_13(size_t COUNT10, uint64_t* COUNT12) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT12, 1);
}
}
__global__ void main_13(size_t COUNT10, DBDecimalType* MAT12aggr0__tmp_attr0, DBI16Type* MAT12nation__n_name_encoded, uint64_t* MAT_IDX12, DBDecimalType* aggr0__tmp_attr0, DBI16Type* nation__n_name_encoded) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize buffers
DBI16Type reg_nation__n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_nation__n_name_encoded[ITEM] = nation__n_name_encoded[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx12 = atomicAdd((int*)MAT_IDX12, 1);
MAT12nation__n_name_encoded[mat_idx12] = reg_nation__n_name_encoded[ITEM];
MAT12aggr0__tmp_attr0[mat_idx12] = reg_aggr0__tmp_attr0[ITEM];
}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map) {
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)region_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT0, d_region__r_name, region_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)region_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_region__r_name, d_region__r_regionkey, region_size);
//Materialize count
uint64_t* d_COUNT2;
hipMalloc(&d_COUNT2, sizeof(uint64_t));
hipMemset(d_COUNT2, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_COUNT2, d_HT_0.ref(cuco::find), d_nation__n_regionkey, nation_size);
uint64_t COUNT2;
hipMemcpy(&COUNT2, d_COUNT2, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 2);
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_BUF_2, d_BUF_IDX_2, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::insert), d_nation__n_nationkey, d_nation__n_regionkey, nation_size);
//Materialize count
uint64_t* d_COUNT4;
hipMalloc(&d_COUNT4, sizeof(uint64_t));
hipMemset(d_COUNT4, 0, sizeof(uint64_t));
count_5<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_2, d_COUNT4, d_HT_2.ref(cuco::find), d_customer__c_nationkey, customer_size);
uint64_t COUNT4;
hipMemcpy(&COUNT4, d_COUNT4, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 3);
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_2, d_BUF_4, d_BUF_IDX_4, d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::insert), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
//Materialize count
uint64_t* d_COUNT6;
hipMalloc(&d_COUNT6, sizeof(uint64_t));
hipMemset(d_COUNT6, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_COUNT6, d_HT_4.ref(cuco::find), d_orders__o_custkey, d_orders__o_orderdate, orders_size);
uint64_t COUNT6;
hipMemcpy(&COUNT6, d_COUNT6, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 4);
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_BUF_6, d_BUF_IDX_6, d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
//Materialize count
uint64_t* d_COUNT8;
hipMalloc(&d_COUNT8, sizeof(uint64_t));
hipMemset(d_COUNT8, 0, sizeof(uint64_t));
count_9<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT8, supplier_size);
uint64_t COUNT8;
hipMemcpy(&COUNT8, d_COUNT8, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_8;
hipMalloc(&d_BUF_IDX_8, sizeof(uint64_t));
hipMemset(d_BUF_IDX_8, 0, sizeof(uint64_t));
uint64_t* d_BUF_8;
hipMalloc(&d_BUF_8, sizeof(uint64_t) * COUNT8 * 1);
auto d_HT_8 = cuco::static_map{ (int)COUNT8*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_9<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_8, d_BUF_IDX_8, d_HT_8.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
//Create aggregation hash table
auto d_HT_10 = cuco::static_map{ (int)22857*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_11<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_6, d_BUF_8, d_HT_10.ref(cuco::insert), d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::find), d_customer__c_nationkey, d_lineitem__l_orderkey, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded);
size_t COUNT10 = d_HT_10.size();
thrust::device_vector<int64_t> keys_10(COUNT10), vals_10(COUNT10);
d_HT_10.retrieve_all(keys_10.begin(), vals_10.begin());
d_HT_10.clear();
int64_t* raw_keys10 = thrust::raw_pointer_cast(keys_10.data());
insertKeys<<<std::ceil((float)COUNT10/128.), 128>>>(raw_keys10, d_HT_10.ref(cuco::insert), COUNT10);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT10);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT10);
DBI16Type* d_KEY_10nation__n_name_encoded;
hipMalloc(&d_KEY_10nation__n_name_encoded, sizeof(DBI16Type) * COUNT10);
hipMemset(d_KEY_10nation__n_name_encoded, 0, sizeof(DBI16Type) * COUNT10);
main_11<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_6, d_BUF_8, d_HT_10.ref(cuco::find), d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::find), d_KEY_10nation__n_name_encoded, d_aggr0__tmp_attr0, d_customer__c_nationkey, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded);
//Materialize count
uint64_t* d_COUNT12;
hipMalloc(&d_COUNT12, sizeof(uint64_t));
hipMemset(d_COUNT12, 0, sizeof(uint64_t));
count_13<<<std::ceil((float)COUNT10/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT10, d_COUNT12);
uint64_t COUNT12;
hipMemcpy(&COUNT12, d_COUNT12, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX12;
hipMalloc(&d_MAT_IDX12, sizeof(uint64_t));
hipMemset(d_MAT_IDX12, 0, sizeof(uint64_t));
auto MAT12nation__n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT12);
DBI16Type* d_MAT12nation__n_name_encoded;
hipMalloc(&d_MAT12nation__n_name_encoded, sizeof(DBI16Type) * COUNT12);
auto MAT12aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT12);
DBDecimalType* d_MAT12aggr0__tmp_attr0;
hipMalloc(&d_MAT12aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT12);
main_13<<<std::ceil((float)COUNT10/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT10, d_MAT12aggr0__tmp_attr0, d_MAT12nation__n_name_encoded, d_MAT_IDX12, d_aggr0__tmp_attr0, d_KEY_10nation__n_name_encoded);
hipMemcpy(MAT12nation__n_name_encoded, d_MAT12nation__n_name_encoded, sizeof(DBI16Type) * COUNT12, hipMemcpyDeviceToHost);
hipMemcpy(MAT12aggr0__tmp_attr0, d_MAT12aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT12, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT12; i++) { std::cout << "" << nation__n_name_map[MAT12nation__n_name_encoded[i]];
std::cout << "," << MAT12aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_COUNT0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_COUNT2);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_COUNT4);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_COUNT6);
hipFree(d_BUF_8);
hipFree(d_BUF_IDX_8);
hipFree(d_COUNT8);
hipFree(d_KEY_10nation__n_name_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT12);
hipFree(d_MAT12aggr0__tmp_attr0);
hipFree(d_MAT12nation__n_name_encoded);
hipFree(d_MAT_IDX12);
free(MAT12aggr0__tmp_attr0);
free(MAT12nation__n_name_encoded);
}