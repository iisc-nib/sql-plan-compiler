#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_orders__o_orderdate[ITEM], 8674, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate[ITEM], 8766, Predicate::lt);
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_orders__o_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_0.insert(cuco::pair{KEY_0[ITEM], ITEM*TB + tid});
BUF_0[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_INSERT HT_2, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_2.insert(cuco::pair{KEY_2[ITEM], ITEM*TB + tid});
BUF_2[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_INSERT HT_4, DBI32Type* nation__n_nationkey, size_t nation_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_nation__n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_nation__n_nationkey[ITEM] = nation__n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_nation__n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_4.insert(cuco::pair{KEY_4[ITEM], ITEM*TB + tid});
BUF_4[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_7(uint64_t* BUF_0, uint64_t* BUF_2, uint64_t* BUF_4, HASHTABLE_PROBE HT_0, HASHTABLE_PROBE HT_2, HASHTABLE_PROBE HT_4, HASHTABLE_FIND HT_6, DBI32Type* KEY_6customer__c_custkey, int* SLOT_COUNT_6, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr__c_acctbal, DBI16Type* aggr__n_name_encoded, DBDecimalType* customer__c_acctbal, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBCharType* lineitem__l_returnflag, size_t lineitem_size, DBI16Type* nation__n_name_encoded, DBI32Type* orders__o_custkey) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBCharType reg_lineitem__l_returnflag[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_returnflag[ITEM] = lineitem__l_returnflag[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_returnflag[ITEM], 'R', Predicate::eq);
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
//Probe Hash table
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_custkey[ITEM] = orders__o_custkey[BUF_0[slot_second0[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_orders__o_custkey[ITEM];
}
//Probe Hash table
int64_t slot_second2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_2 = HT_2.find(KEY_2[ITEM]);
if (SLOT_2 == HT_2.end()) {selection_flags[ITEM] = 0; continue;}
slot_second2[ITEM] = SLOT_2->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[BUF_2[slot_second2[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_customer__c_nationkey[ITEM];
}
//Probe Hash table
int64_t slot_second4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0; continue;}
slot_second4[ITEM] = SLOT_4->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_custkey[ITEM] = customer__c_custkey[BUF_2[slot_second2[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_customer__c_custkey[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]));
}
DBI16Type reg_nation__n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_nation__n_name_encoded[ITEM] = nation__n_name_encoded[BUF_4[slot_second4[ITEM] * 1 + 0]];
}
DBDecimalType reg_customer__c_acctbal[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_acctbal[ITEM] = customer__c_acctbal[BUF_2[slot_second2[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_6 = get_aggregation_slot(KEY_6[ITEM], HT_6, SLOT_COUNT_6);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_6], reg_map0__tmp_attr1[ITEM]);
aggregate_any(&aggr__n_name_encoded[buf_idx_6], reg_nation__n_name_encoded[ITEM]);
aggregate_any(&aggr__c_acctbal[buf_idx_6], reg_customer__c_acctbal[ITEM]);
KEY_6customer__c_custkey[buf_idx_6] = reg_customer__c_custkey[ITEM];
}
}
__global__ void main_9(size_t COUNT6, DBDecimalType* MAT8aggr0__tmp_attr0, DBDecimalType* MAT8aggr__c_acctbal, DBI16Type* MAT8aggr__n_name_encoded, DBI32Type* MAT8customer__c_custkey, uint64_t* MAT_IDX8, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr__c_acctbal, DBI16Type* aggr__n_name_encoded, DBI32Type* customer__c_custkey) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize buffers
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT6); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT6); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
DBDecimalType reg_aggr__c_acctbal[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT6); ++ITEM) {
reg_aggr__c_acctbal[ITEM] = aggr__c_acctbal[ITEM*TB + tid];
}
DBI16Type reg_aggr__n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT6); ++ITEM) {
reg_aggr__n_name_encoded[ITEM] = aggr__n_name_encoded[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT6); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx8 = atomicAdd((int*)MAT_IDX8, 1);
MAT8customer__c_custkey[mat_idx8] = reg_customer__c_custkey[ITEM];
MAT8aggr0__tmp_attr0[mat_idx8] = reg_aggr0__tmp_attr0[ITEM];
MAT8aggr__c_acctbal[mat_idx8] = reg_aggr__c_acctbal[ITEM];
MAT8aggr__n_name_encoded[mat_idx8] = reg_aggr__n_name_encoded[ITEM];
}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t COUNT0 = orders_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
size_t COUNT2 = customer_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 1);
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_2, d_BUF_IDX_2, d_HT_2.ref(cuco::insert), d_customer__c_custkey, customer_size);
size_t COUNT4 = nation_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 1);
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_BUF_IDX_4, d_HT_4.ref(cuco::insert), d_nation__n_nationkey, nation_size);
size_t COUNT6 = 45145;
auto d_HT_6 = cuco::static_map{ (int)45145*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_6;
hipMalloc(&d_SLOT_COUNT_6, sizeof(int));
hipMemset(d_SLOT_COUNT_6, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT6);
DBI16Type* d_aggr__n_name_encoded;
hipMalloc(&d_aggr__n_name_encoded, sizeof(DBI16Type) * COUNT6);
hipMemset(d_aggr__n_name_encoded, 0, sizeof(DBI16Type) * COUNT6);
auto aggr__n_name_map = nation__n_name_map;
DBDecimalType* d_aggr__c_acctbal;
hipMalloc(&d_aggr__c_acctbal, sizeof(DBDecimalType) * COUNT6);
hipMemset(d_aggr__c_acctbal, 0, sizeof(DBDecimalType) * COUNT6);
DBI32Type* d_KEY_6customer__c_custkey;
hipMalloc(&d_KEY_6customer__c_custkey, sizeof(DBI32Type) * COUNT6);
hipMemset(d_KEY_6customer__c_custkey, 0, sizeof(DBI32Type) * COUNT6);
main_7<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_BUF_2, d_BUF_4, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::insert_and_find), d_KEY_6customer__c_custkey, d_SLOT_COUNT_6, d_aggr0__tmp_attr0, d_aggr__c_acctbal, d_aggr__n_name_encoded, d_customer__c_acctbal, d_customer__c_custkey, d_customer__c_nationkey, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_returnflag, lineitem_size, d_nation__n_name_encoded, d_orders__o_custkey);
COUNT6 = d_HT_6.size();
size_t COUNT8 = COUNT6;
//Materialize buffers
uint64_t* d_MAT_IDX8;
hipMalloc(&d_MAT_IDX8, sizeof(uint64_t));
hipMemset(d_MAT_IDX8, 0, sizeof(uint64_t));
auto MAT8customer__c_custkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT8);
DBI32Type* d_MAT8customer__c_custkey;
hipMalloc(&d_MAT8customer__c_custkey, sizeof(DBI32Type) * COUNT8);
auto MAT8aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT8);
DBDecimalType* d_MAT8aggr0__tmp_attr0;
hipMalloc(&d_MAT8aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT8);
auto MAT8aggr__c_acctbal = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT8);
DBDecimalType* d_MAT8aggr__c_acctbal;
hipMalloc(&d_MAT8aggr__c_acctbal, sizeof(DBDecimalType) * COUNT8);
auto MAT8aggr__n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT8);
DBI16Type* d_MAT8aggr__n_name_encoded;
hipMalloc(&d_MAT8aggr__n_name_encoded, sizeof(DBI16Type) * COUNT8);
main_9<<<std::ceil((float)COUNT6/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT6, d_MAT8aggr0__tmp_attr0, d_MAT8aggr__c_acctbal, d_MAT8aggr__n_name_encoded, d_MAT8customer__c_custkey, d_MAT_IDX8, d_aggr0__tmp_attr0, d_aggr__c_acctbal, d_aggr__n_name_encoded, d_KEY_6customer__c_custkey);
uint64_t MATCOUNT_8 = 0;
hipMemcpy(&MATCOUNT_8, d_MAT_IDX8, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT8customer__c_custkey, d_MAT8customer__c_custkey, sizeof(DBI32Type) * COUNT8, hipMemcpyDeviceToHost);
hipMemcpy(MAT8aggr0__tmp_attr0, d_MAT8aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT8, hipMemcpyDeviceToHost);
hipMemcpy(MAT8aggr__c_acctbal, d_MAT8aggr__c_acctbal, sizeof(DBDecimalType) * COUNT8, hipMemcpyDeviceToHost);
hipMemcpy(MAT8aggr__n_name_encoded, d_MAT8aggr__n_name_encoded, sizeof(DBI16Type) * COUNT8, hipMemcpyDeviceToHost);
for (auto i=0ull; i < MATCOUNT_8; i++) { std::cout << "" << MAT8customer__c_custkey[i];
std::cout << "|" << MAT8aggr0__tmp_attr0[i];
std::cout << "|" << MAT8aggr__c_acctbal[i];
std::cout << "|" << aggr__n_name_map[MAT8aggr__n_name_encoded[i]];
std::cout << std::endl; }
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_KEY_6customer__c_custkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr__c_acctbal);
hipFree(d_aggr__n_name_encoded);
hipFree(d_MAT8aggr0__tmp_attr0);
hipFree(d_MAT8aggr__c_acctbal);
hipFree(d_MAT8aggr__n_name_encoded);
hipFree(d_MAT8customer__c_custkey);
hipFree(d_MAT_IDX8);
free(MAT8aggr0__tmp_attr0);
free(MAT8aggr__c_acctbal);
free(MAT8aggr__n_name_encoded);
free(MAT8customer__c_custkey);
}