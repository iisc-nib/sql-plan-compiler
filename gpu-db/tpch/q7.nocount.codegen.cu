#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBStringType* n1___n_name, DBI32Type* n1___n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n1___n_name = n1___n_name[tid];
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
uint64_t KEY_0 = 0;
auto reg_n1___n_nationkey = n1___n_nationkey[tid];

KEY_0 |= reg_n1___n_nationkey;
// Insert hash table kernel;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0, buf_idx_0});
BUF_0[buf_idx_0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_INSERT HT_2, DBStringType* n2___n_name, DBI32Type* n2___n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n2___n_name = n2___n_name[tid];
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
uint64_t KEY_2 = 0;
auto reg_n2___n_nationkey = n2___n_nationkey[tid];

KEY_2 |= reg_n2___n_nationkey;
// Insert hash table kernel;
auto buf_idx_2 = atomicAdd((int*)BUF_IDX_2, 1);
HT_2.insert(cuco::pair{KEY_2, buf_idx_2});
BUF_2[buf_idx_2 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_2, uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_PROBE HT_2, HASHTABLE_INSERT HT_4, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_2 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_2 |= reg_customer__c_nationkey;
//Probe Hash table
HT_2.for_each(KEY_2, [&] __device__ (auto const SLOT_2) {
auto const [slot_first2, slot_second2] = SLOT_2;
if (!(true)) return;
uint64_t KEY_4 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_4 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_4 = atomicAdd((int*)BUF_IDX_4, 1);
HT_4.insert(cuco::pair{KEY_4, buf_idx_4});
BUF_4[buf_idx_4 * 2 + 0] = BUF_2[slot_second2 * 1 + 0];
BUF_4[buf_idx_4 * 2 + 1] = tid;
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_7(uint64_t* BUF_4, uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_PROBE HT_4, HASHTABLE_INSERT HT_6, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_4 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_4 |= reg_orders__o_custkey;
//Probe Hash table
HT_4.for_each(KEY_4, [&] __device__ (auto const SLOT_4) {
auto const [slot_first4, slot_second4] = SLOT_4;
if (!(true)) return;
uint64_t KEY_6 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_6 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_6 = atomicAdd((int*)BUF_IDX_6, 1);
HT_6.insert(cuco::pair{KEY_6, buf_idx_6});
BUF_6[buf_idx_6 * 3 + 0] = tid;
BUF_6[buf_idx_6 * 3 + 1] = BUF_4[slot_second4 * 2 + 0];
BUF_6[buf_idx_6 * 3 + 2] = BUF_4[slot_second4 * 2 + 1];
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_9(uint64_t* BUF_0, uint64_t* BUF_8, uint64_t* BUF_IDX_8, HASHTABLE_PROBE HT_0, HASHTABLE_INSERT HT_8, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_0 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_0 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_0.for_each(KEY_0, [&] __device__ (auto const SLOT_0) {
auto const [slot_first0, slot_second0] = SLOT_0;
if (!(true)) return;
uint64_t KEY_8 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_8 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_8 = atomicAdd((int*)BUF_IDX_8, 1);
HT_8.insert(cuco::pair{KEY_8, buf_idx_8});
BUF_8[buf_idx_8 * 2 + 0] = tid;
BUF_8[buf_idx_8 * 2 + 1] = BUF_0[slot_second0 * 1 + 0];
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_11(uint64_t* BUF_6, uint64_t* BUF_8, HASHTABLE_FIND HT_10, HASHTABLE_PROBE HT_6, HASHTABLE_PROBE HT_8, DBI64Type* KEY_10map0__tmp_attr0, DBI16Type* KEY_10n1___n_name_encoded, DBI16Type* KEY_10n2___n_name_encoded, int* SLOT_COUNT_10, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n1___n_name, DBI16Type* n1___n_name_encoded, DBStringType* n2___n_name, DBI16Type* n2___n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9861, Predicate::lte))) return;
uint64_t KEY_6 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_6 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_6.for_each(KEY_6, [&] __device__ (auto const SLOT_6) {
auto const [slot_first6, slot_second6] = SLOT_6;
if (!(true)) return;
uint64_t KEY_8 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_8 |= reg_lineitem__l_suppkey;
//Probe Hash table
HT_8.for_each(KEY_8, [&] __device__ (auto const SLOT_8) {
auto const [slot_first8, slot_second8] = SLOT_8;
auto reg_n1___n_name = n1___n_name[BUF_8[slot_second8 * 2 + 1]];
auto reg_n2___n_name = n2___n_name[BUF_6[slot_second6 * 3 + 1]];
if (!((((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq))) || ((evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) && (true))) return;
uint64_t KEY_10 = 0;
auto reg_n1___n_name_encoded = n1___n_name_encoded[BUF_8[slot_second8 * 2 + 1]];

KEY_10 |= reg_n1___n_name_encoded;
auto reg_n2___n_name_encoded = n2___n_name_encoded[BUF_6[slot_second6 * 3 + 1]];
KEY_10 <<= 16;
KEY_10 |= reg_n2___n_name_encoded;
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_lineitem__l_shipdate);
KEY_10 <<= 32;
KEY_10 |= (DBI32Type)reg_map0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_10 = get_aggregation_slot(KEY_10, HT_10, SLOT_COUNT_10);
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_10], reg_map0__tmp_attr1);
KEY_10n1___n_name_encoded[buf_idx_10] = reg_n1___n_name_encoded;
KEY_10n2___n_name_encoded[buf_idx_10] = reg_n2___n_name_encoded;
KEY_10map0__tmp_attr0[buf_idx_10] = reg_map0__tmp_attr0;
});
});
}
__global__ void main_13(size_t COUNT10, DBDecimalType* MAT12aggr0__tmp_attr2, DBI64Type* MAT12map0__tmp_attr0, DBI16Type* MAT12n1___n_name_encoded, DBI16Type* MAT12n2___n_name_encoded, uint64_t* MAT_IDX12, DBDecimalType* aggr0__tmp_attr2, DBI64Type* map0__tmp_attr0, DBI16Type* n1___n_name_encoded, DBI16Type* n2___n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT10) return;
//Materialize buffers
auto mat_idx12 = atomicAdd((int*)MAT_IDX12, 1);
auto reg_n1___n_name_encoded = n1___n_name_encoded[tid];
MAT12n1___n_name_encoded[mat_idx12] = reg_n1___n_name_encoded;
auto reg_n2___n_name_encoded = n2___n_name_encoded[tid];
MAT12n2___n_name_encoded[mat_idx12] = reg_n2___n_name_encoded;
auto reg_map0__tmp_attr0 = map0__tmp_attr0[tid];
MAT12map0__tmp_attr0[mat_idx12] = reg_map0__tmp_attr0;
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT12aggr0__tmp_attr2[mat_idx12] = reg_aggr0__tmp_attr2;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map) {
size_t COUNT0 = nation_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::experimental::static_multimap{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)nation_size/128.), 128>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
hipFree(d_BUF_IDX_0);
size_t COUNT2 = nation_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 1);
auto d_HT_2 = cuco::experimental::static_multimap{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)nation_size/128.), 128>>>(d_BUF_2, d_BUF_IDX_2, d_HT_2.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
hipFree(d_BUF_IDX_2);
size_t COUNT4 = customer_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 2);
auto d_HT_4 = cuco::experimental::static_multimap{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_2, d_BUF_4, d_BUF_IDX_4, d_HT_2.ref(cuco::for_each), d_HT_4.ref(cuco::insert), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
hipFree(d_BUF_IDX_4);
size_t COUNT6 = orders_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 3);
auto d_HT_6 = cuco::experimental::static_multimap{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)orders_size/128.), 128>>>(d_BUF_4, d_BUF_6, d_BUF_IDX_6, d_HT_4.ref(cuco::for_each), d_HT_6.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderkey, orders_size);
hipFree(d_BUF_IDX_6);
size_t COUNT8 = supplier_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_8;
hipMalloc(&d_BUF_IDX_8, sizeof(uint64_t));
hipMemset(d_BUF_IDX_8, 0, sizeof(uint64_t));
uint64_t* d_BUF_8;
hipMalloc(&d_BUF_8, sizeof(uint64_t) * COUNT8 * 2);
auto d_HT_8 = cuco::experimental::static_multimap{ (int)COUNT8*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_9<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_0, d_BUF_8, d_BUF_IDX_8, d_HT_0.ref(cuco::for_each), d_HT_8.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
hipFree(d_BUF_IDX_8);
size_t COUNT10 = 13634;
auto d_HT_10 = cuco::static_map{ (int)13634*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_10;
hipMalloc(&d_SLOT_COUNT_10, sizeof(int));
hipMemset(d_SLOT_COUNT_10, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT10);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT10);
DBI16Type* d_KEY_10n1___n_name_encoded;
hipMalloc(&d_KEY_10n1___n_name_encoded, sizeof(DBI16Type) * COUNT10);
hipMemset(d_KEY_10n1___n_name_encoded, 0, sizeof(DBI16Type) * COUNT10);
DBI16Type* d_KEY_10n2___n_name_encoded;
hipMalloc(&d_KEY_10n2___n_name_encoded, sizeof(DBI16Type) * COUNT10);
hipMemset(d_KEY_10n2___n_name_encoded, 0, sizeof(DBI16Type) * COUNT10);
DBI64Type* d_KEY_10map0__tmp_attr0;
hipMalloc(&d_KEY_10map0__tmp_attr0, sizeof(DBI64Type) * COUNT10);
hipMemset(d_KEY_10map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT10);
main_11<<<std::ceil((float)lineitem_size/128.), 128>>>(d_BUF_6, d_BUF_8, d_HT_10.ref(cuco::insert_and_find), d_HT_6.ref(cuco::for_each), d_HT_8.ref(cuco::for_each), d_KEY_10map0__tmp_attr0, d_KEY_10n1___n_name_encoded, d_KEY_10n2___n_name_encoded, d_SLOT_COUNT_10, d_aggr0__tmp_attr2, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_nation__n_name_encoded, d_nation__n_name, d_nation__n_name_encoded);
COUNT10 = d_HT_10.size();
size_t COUNT12 = COUNT10;
//Materialize buffers
uint64_t* d_MAT_IDX12;
hipMalloc(&d_MAT_IDX12, sizeof(uint64_t));
hipMemset(d_MAT_IDX12, 0, sizeof(uint64_t));
auto MAT12n1___n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT12);
DBI16Type* d_MAT12n1___n_name_encoded;
hipMalloc(&d_MAT12n1___n_name_encoded, sizeof(DBI16Type) * COUNT12);
auto MAT12n2___n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT12);
DBI16Type* d_MAT12n2___n_name_encoded;
hipMalloc(&d_MAT12n2___n_name_encoded, sizeof(DBI16Type) * COUNT12);
auto MAT12map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT12);
DBI64Type* d_MAT12map0__tmp_attr0;
hipMalloc(&d_MAT12map0__tmp_attr0, sizeof(DBI64Type) * COUNT12);
auto MAT12aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT12);
DBDecimalType* d_MAT12aggr0__tmp_attr2;
hipMalloc(&d_MAT12aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT12);
main_13<<<std::ceil((float)COUNT10/128.), 128>>>(COUNT10, d_MAT12aggr0__tmp_attr2, d_MAT12map0__tmp_attr0, d_MAT12n1___n_name_encoded, d_MAT12n2___n_name_encoded, d_MAT_IDX12, d_aggr0__tmp_attr2, d_KEY_10map0__tmp_attr0, d_KEY_10n1___n_name_encoded, d_KEY_10n2___n_name_encoded);
hipMemcpy(MAT12n1___n_name_encoded, d_MAT12n1___n_name_encoded, sizeof(DBI16Type) * COUNT12, hipMemcpyDeviceToHost);
hipMemcpy(MAT12n2___n_name_encoded, d_MAT12n2___n_name_encoded, sizeof(DBI16Type) * COUNT12, hipMemcpyDeviceToHost);
hipMemcpy(MAT12map0__tmp_attr0, d_MAT12map0__tmp_attr0, sizeof(DBI64Type) * COUNT12, hipMemcpyDeviceToHost);
hipMemcpy(MAT12aggr0__tmp_attr2, d_MAT12aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT12, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT12; i++) { std::cout << "" << n1___n_name_map[MAT12n1___n_name_encoded[i]];
std::cout << "," << n2___n_name_map[MAT12n2___n_name_encoded[i]];
std::cout << "," << MAT12map0__tmp_attr0[i];
std::cout << "," << MAT12aggr0__tmp_attr2[i];
std::cout << std::endl; }
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_BUF_8);
hipFree(d_BUF_IDX_8);
hipFree(d_KEY_10map0__tmp_attr0);
hipFree(d_KEY_10n1___n_name_encoded);
hipFree(d_KEY_10n2___n_name_encoded);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_MAT12aggr0__tmp_attr2);
hipFree(d_MAT12map0__tmp_attr0);
hipFree(d_MAT12n1___n_name_encoded);
hipFree(d_MAT12n2___n_name_encoded);
hipFree(d_MAT_IDX12);
free(MAT12aggr0__tmp_attr2);
free(MAT12map0__tmp_attr0);
free(MAT12n1___n_name_encoded);
free(MAT12n2___n_name_encoded);
}