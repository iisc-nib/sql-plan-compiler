#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBStringType* n1___n_name, DBI32Type* n1___n_nationkey, size_t nation_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_n1___n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n1___n_name[ITEM] = n1___n_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n1___n_name[ITEM], "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name[ITEM], "GERMANY", Predicate::eq));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n1___n_name[ITEM], "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name[ITEM], "GERMANY", Predicate::eq));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n1___n_name[ITEM], "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name[ITEM], "GERMANY", Predicate::eq));
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_n1___n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n1___n_nationkey[ITEM] = n1___n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_n1___n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_0.insert(cuco::pair{KEY_0[ITEM], ITEM*TB + tid});
BUF_0[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_INSERT HT_2, DBStringType* n2___n_name, DBI32Type* n2___n_nationkey, size_t nation_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_n2___n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n2___n_name[ITEM] = n2___n_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n2___n_name[ITEM], "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name[ITEM], "FRANCE", Predicate::eq));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n2___n_name[ITEM], "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name[ITEM], "FRANCE", Predicate::eq));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n2___n_name[ITEM], "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name[ITEM], "FRANCE", Predicate::eq));
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_n2___n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n2___n_nationkey[ITEM] = n2___n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_n2___n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_2.insert(cuco::pair{KEY_2[ITEM], ITEM*TB + tid});
BUF_2[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_2, uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_PROBE HT_2, HASHTABLE_INSERT HT_4, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_customer__c_nationkey[ITEM];
}
//Probe Hash table
int64_t slot_second2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_2 = HT_2.find(KEY_2[ITEM]);
if (SLOT_2 == HT_2.end()) {selection_flags[ITEM] = 0; continue;}
slot_second2[ITEM] = SLOT_2->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_4.insert(cuco::pair{KEY_4[ITEM], ITEM*TB + tid});
BUF_4[(ITEM*TB + tid) * 2 + 0] = BUF_2[slot_second2[ITEM] * 1 + 0];
BUF_4[(ITEM*TB + tid) * 2 + 1] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_7(uint64_t* BUF_4, uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_PROBE HT_4, HASHTABLE_INSERT HT_6, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_custkey[ITEM] = orders__o_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_orders__o_custkey[ITEM];
}
//Probe Hash table
int64_t slot_second4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0; continue;}
slot_second4[ITEM] = SLOT_4->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_orders__o_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_6.insert(cuco::pair{KEY_6[ITEM], ITEM*TB + tid});
BUF_6[(ITEM*TB + tid) * 3 + 0] = ITEM*TB + tid;
BUF_6[(ITEM*TB + tid) * 3 + 1] = BUF_4[slot_second4[ITEM] * 2 + 0];
BUF_6[(ITEM*TB + tid) * 3 + 2] = BUF_4[slot_second4[ITEM] * 2 + 1];
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_9(uint64_t* BUF_0, uint64_t* BUF_8, uint64_t* BUF_IDX_8, HASHTABLE_PROBE HT_0, HASHTABLE_INSERT HT_8, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_nationkey[ITEM] = supplier__s_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_supplier__s_nationkey[ITEM];
}
//Probe Hash table
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_8.insert(cuco::pair{KEY_8[ITEM], ITEM*TB + tid});
BUF_8[(ITEM*TB + tid) * 2 + 0] = ITEM*TB + tid;
BUF_8[(ITEM*TB + tid) * 2 + 1] = BUF_0[slot_second0[ITEM] * 1 + 0];
}
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_11(uint64_t* BUF_6, uint64_t* BUF_8, HASHTABLE_FIND HT_10, HASHTABLE_PROBE HT_6, HASHTABLE_PROBE HT_8, DBI64Type* KEY_10map0__tmp_attr0, DBI16Type* KEY_10n1___n_name_encoded, DBI16Type* KEY_10n2___n_name_encoded, int* SLOT_COUNT_10, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n1___n_name, DBI16Type* n1___n_name_encoded, DBStringType* n2___n_name, DBI16Type* n2___n_name_encoded) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_lineitem__l_shipdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipdate[ITEM] = lineitem__l_shipdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 9131, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 9861, Predicate::lte);
}
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
//Probe Hash table
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_suppkey[ITEM] = lineitem__l_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_lineitem__l_suppkey[ITEM];
}
//Probe Hash table
int64_t slot_second8[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_8 = HT_8.find(KEY_8[ITEM]);
if (SLOT_8 == HT_8.end()) {selection_flags[ITEM] = 0; continue;}
slot_second8[ITEM] = SLOT_8->second;
}
DBStringType reg_n1___n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_n1___n_name[ITEM] = n1___n_name[BUF_8[slot_second8[ITEM] * 2 + 1]];
}
DBStringType reg_n2___n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_n2___n_name[ITEM] = n2___n_name[BUF_6[slot_second6[ITEM] * 3 + 1]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (((evaluatePredicate(reg_n1___n_name[ITEM], "FRANCE", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name[ITEM], "GERMANY", Predicate::eq))) || ((evaluatePredicate(reg_n1___n_name[ITEM], "GERMANY", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name[ITEM], "FRANCE", Predicate::eq)))) && (true);
}
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI16Type reg_n1___n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_n1___n_name_encoded[ITEM] = n1___n_name_encoded[BUF_8[slot_second8[ITEM] * 2 + 1]];
}
DBI16Type reg_n2___n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_n2___n_name_encoded[ITEM] = n2___n_name_encoded[BUF_6[slot_second6[ITEM] * 3 + 1]];
}
DBI64Type reg_map0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr0[ITEM] = ExtractFromDate("year", reg_lineitem__l_shipdate[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_n1___n_name_encoded[ITEM];
KEY_10[ITEM] <<= 16;
KEY_10[ITEM] |= reg_n2___n_name_encoded[ITEM];
KEY_10[ITEM] <<= 32;
KEY_10[ITEM] |= (DBI32Type)reg_map0__tmp_attr0[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_10 = get_aggregation_slot(KEY_10[ITEM], HT_10, SLOT_COUNT_10);
aggregate_sum(&aggr0__tmp_attr2[buf_idx_10], reg_map0__tmp_attr1[ITEM]);
KEY_10n1___n_name_encoded[buf_idx_10] = reg_n1___n_name_encoded[ITEM];
KEY_10n2___n_name_encoded[buf_idx_10] = reg_n2___n_name_encoded[ITEM];
KEY_10map0__tmp_attr0[buf_idx_10] = reg_map0__tmp_attr0[ITEM];
}
}
__global__ void main_13(size_t COUNT10, DBDecimalType* MAT12aggr0__tmp_attr2, DBI64Type* MAT12map0__tmp_attr0, DBI16Type* MAT12n1___n_name_encoded, DBI16Type* MAT12n2___n_name_encoded, uint64_t* MAT_IDX12, DBDecimalType* aggr0__tmp_attr2, DBI64Type* map0__tmp_attr0, DBI16Type* n1___n_name_encoded, DBI16Type* n2___n_name_encoded) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize buffers
DBI16Type reg_n1___n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_n1___n_name_encoded[ITEM] = n1___n_name_encoded[ITEM*TB + tid];
}
DBI16Type reg_n2___n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_n2___n_name_encoded[ITEM] = n2___n_name_encoded[ITEM*TB + tid];
}
DBI64Type reg_map0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_map0__tmp_attr0[ITEM] = map0__tmp_attr0[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_aggr0__tmp_attr2[ITEM] = aggr0__tmp_attr2[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx12 = atomicAdd((int*)MAT_IDX12, 1);
MAT12n1___n_name_encoded[mat_idx12] = reg_n1___n_name_encoded[ITEM];
MAT12n2___n_name_encoded[mat_idx12] = reg_n2___n_name_encoded[ITEM];
MAT12map0__tmp_attr0[mat_idx12] = reg_map0__tmp_attr0[ITEM];
MAT12aggr0__tmp_attr2[mat_idx12] = reg_aggr0__tmp_attr2[ITEM];
}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t COUNT0 = nation_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
size_t COUNT2 = nation_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 1);
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_2, d_BUF_IDX_2, d_HT_2.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
size_t COUNT4 = customer_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 2);
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_2, d_BUF_4, d_BUF_IDX_4, d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::insert), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
size_t COUNT6 = orders_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 3);
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_BUF_6, d_BUF_IDX_6, d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderkey, orders_size);
size_t COUNT8 = supplier_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_8;
hipMalloc(&d_BUF_IDX_8, sizeof(uint64_t));
hipMemset(d_BUF_IDX_8, 0, sizeof(uint64_t));
uint64_t* d_BUF_8;
hipMalloc(&d_BUF_8, sizeof(uint64_t) * COUNT8 * 2);
auto d_HT_8 = cuco::static_map{ (int)COUNT8*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_9<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_BUF_8, d_BUF_IDX_8, d_HT_0.ref(cuco::find), d_HT_8.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
size_t COUNT10 = 13634;
auto d_HT_10 = cuco::static_map{ (int)13634*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_10;
hipMalloc(&d_SLOT_COUNT_10, sizeof(int));
hipMemset(d_SLOT_COUNT_10, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT10);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT10);
DBI16Type* d_KEY_10n1___n_name_encoded;
hipMalloc(&d_KEY_10n1___n_name_encoded, sizeof(DBI16Type) * COUNT10);
hipMemset(d_KEY_10n1___n_name_encoded, 0, sizeof(DBI16Type) * COUNT10);
DBI16Type* d_KEY_10n2___n_name_encoded;
hipMalloc(&d_KEY_10n2___n_name_encoded, sizeof(DBI16Type) * COUNT10);
hipMemset(d_KEY_10n2___n_name_encoded, 0, sizeof(DBI16Type) * COUNT10);
DBI64Type* d_KEY_10map0__tmp_attr0;
hipMalloc(&d_KEY_10map0__tmp_attr0, sizeof(DBI64Type) * COUNT10);
hipMemset(d_KEY_10map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT10);
main_11<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_6, d_BUF_8, d_HT_10.ref(cuco::insert_and_find), d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::find), d_KEY_10map0__tmp_attr0, d_KEY_10n1___n_name_encoded, d_KEY_10n2___n_name_encoded, d_SLOT_COUNT_10, d_aggr0__tmp_attr2, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_nation__n_name_encoded, d_nation__n_name, d_nation__n_name_encoded);
COUNT10 = d_HT_10.size();
size_t COUNT12 = COUNT10;
//Materialize buffers
uint64_t* d_MAT_IDX12;
hipMalloc(&d_MAT_IDX12, sizeof(uint64_t));
hipMemset(d_MAT_IDX12, 0, sizeof(uint64_t));
auto MAT12n1___n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT12);
DBI16Type* d_MAT12n1___n_name_encoded;
hipMalloc(&d_MAT12n1___n_name_encoded, sizeof(DBI16Type) * COUNT12);
auto MAT12n2___n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT12);
DBI16Type* d_MAT12n2___n_name_encoded;
hipMalloc(&d_MAT12n2___n_name_encoded, sizeof(DBI16Type) * COUNT12);
auto MAT12map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT12);
DBI64Type* d_MAT12map0__tmp_attr0;
hipMalloc(&d_MAT12map0__tmp_attr0, sizeof(DBI64Type) * COUNT12);
auto MAT12aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT12);
DBDecimalType* d_MAT12aggr0__tmp_attr2;
hipMalloc(&d_MAT12aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT12);
main_13<<<std::ceil((float)COUNT10/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT10, d_MAT12aggr0__tmp_attr2, d_MAT12map0__tmp_attr0, d_MAT12n1___n_name_encoded, d_MAT12n2___n_name_encoded, d_MAT_IDX12, d_aggr0__tmp_attr2, d_KEY_10map0__tmp_attr0, d_KEY_10n1___n_name_encoded, d_KEY_10n2___n_name_encoded);
uint64_t MATCOUNT_12 = 0;
hipMemcpy(&MATCOUNT_12, d_MAT_IDX12, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT12n1___n_name_encoded, d_MAT12n1___n_name_encoded, sizeof(DBI16Type) * COUNT12, hipMemcpyDeviceToHost);
hipMemcpy(MAT12n2___n_name_encoded, d_MAT12n2___n_name_encoded, sizeof(DBI16Type) * COUNT12, hipMemcpyDeviceToHost);
hipMemcpy(MAT12map0__tmp_attr0, d_MAT12map0__tmp_attr0, sizeof(DBI64Type) * COUNT12, hipMemcpyDeviceToHost);
hipMemcpy(MAT12aggr0__tmp_attr2, d_MAT12aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT12, hipMemcpyDeviceToHost);
for (auto i=0ull; i < MATCOUNT_12; i++) { std::cout << "" << n1___n_name_map[MAT12n1___n_name_encoded[i]];
std::cout << "|" << n2___n_name_map[MAT12n2___n_name_encoded[i]];
std::cout << "|" << MAT12map0__tmp_attr0[i];
std::cout << "|" << MAT12aggr0__tmp_attr2[i];
std::cout << std::endl; }
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_BUF_8);
hipFree(d_BUF_IDX_8);
hipFree(d_KEY_10map0__tmp_attr0);
hipFree(d_KEY_10n1___n_name_encoded);
hipFree(d_KEY_10n2___n_name_encoded);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_MAT12aggr0__tmp_attr2);
hipFree(d_MAT12map0__tmp_attr0);
hipFree(d_MAT12n1___n_name_encoded);
hipFree(d_MAT12n2___n_name_encoded);
hipFree(d_MAT_IDX12);
free(MAT12aggr0__tmp_attr2);
free(MAT12map0__tmp_attr0);
free(MAT12n1___n_name_encoded);
free(MAT12n2___n_name_encoded);
}