#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
template<typename HASHTABLE_FIND>
__global__ void main_1(HASHTABLE_FIND HT_0, int* SLOT_COUNT_0, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBDecimalType* lineitem__l_quantity, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::lt))) return;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
if (!(evaluatePredicate(reg_lineitem__l_discount, 0.05, Predicate::gte) && evaluatePredicate(reg_lineitem__l_discount, 0.07, Predicate::lte))) return;
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
if (!(evaluatePredicate(reg_lineitem__l_quantity, 24.0, Predicate::lt))) return;
uint64_t KEY_0 = 0;
//Aggregate in hashtable
auto buf_idx_0 = get_aggregation_slot(KEY_0, HT_0, SLOT_COUNT_0);
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * (reg_lineitem__l_discount);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_0], reg_map0__tmp_attr1);
}
__global__ void main_3(size_t COUNT0, DBDecimalType* MAT2aggr0__tmp_attr0, uint64_t* MAT_IDX2, DBDecimalType* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT0) return;
//Materialize buffers
auto mat_idx2 = atomicAdd((int*)MAT_IDX2, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT2aggr0__tmp_attr0[mat_idx2] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map) {
size_t COUNT0 = 1;
auto d_HT_0 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_0;
hipMalloc(&d_SLOT_COUNT_0, sizeof(int));
hipMemset(d_SLOT_COUNT_0, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT0);
main_1<<<std::ceil((float)lineitem_size/128.), 128>>>(d_HT_0.ref(cuco::insert_and_find), d_SLOT_COUNT_0, d_aggr0__tmp_attr0, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_quantity, d_lineitem__l_shipdate, lineitem_size);
COUNT0 = d_HT_0.size();
size_t COUNT2 = COUNT0;
//Materialize buffers
uint64_t* d_MAT_IDX2;
hipMalloc(&d_MAT_IDX2, sizeof(uint64_t));
hipMemset(d_MAT_IDX2, 0, sizeof(uint64_t));
auto MAT2aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT2);
DBDecimalType* d_MAT2aggr0__tmp_attr0;
hipMalloc(&d_MAT2aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT2);
main_3<<<std::ceil((float)COUNT0/128.), 128>>>(COUNT0, d_MAT2aggr0__tmp_attr0, d_MAT_IDX2, d_aggr0__tmp_attr0);
hipMemcpy(MAT2aggr0__tmp_attr0, d_MAT2aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT2, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT2; i++) { std::cout << "" << MAT2aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_aggr0__tmp_attr0);
hipFree(d_MAT2aggr0__tmp_attr0);
hipFree(d_MAT_IDX2);
free(MAT2aggr0__tmp_attr0);
}