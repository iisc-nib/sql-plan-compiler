#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5b9a27576900(uint64_t* COUNT5b9a27565c70, DBStringType* region__r_name, size_t region_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= region_size) return;
auto reg_region__r_name = region__r_name[tid];
if (!(evaluatePredicate(reg_region__r_name, "ASIA", Predicate::eq))) return;
//Materialize count
atomicAdd((int*)COUNT5b9a27565c70, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5b9a27576900(uint64_t* BUF_5b9a27565c70, uint64_t* BUF_IDX_5b9a27565c70, HASHTABLE_INSERT HT_5b9a27565c70, DBStringType* region__r_name, DBI32Type* region__r_regionkey, size_t region_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= region_size) return;
auto reg_region__r_name = region__r_name[tid];
if (!(evaluatePredicate(reg_region__r_name, "ASIA", Predicate::eq))) return;
uint64_t KEY_5b9a27565c70 = 0;
auto reg_region__r_regionkey = region__r_regionkey[tid];

KEY_5b9a27565c70 |= reg_region__r_regionkey;
// Insert hash table kernel;
auto buf_idx_5b9a27565c70 = atomicAdd((int*)BUF_IDX_5b9a27565c70, 1);
HT_5b9a27565c70.insert(cuco::pair{KEY_5b9a27565c70, buf_idx_5b9a27565c70});
BUF_5b9a27565c70[buf_idx_5b9a27565c70 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5b9a27577ac0(uint64_t* BUF_5b9a27565c70, uint64_t* COUNT5b9a27565ff0, HASHTABLE_PROBE HT_5b9a27565c70, DBI32Type* nation__n_regionkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_5b9a27565c70 = 0;
auto reg_nation__n_regionkey = nation__n_regionkey[tid];

KEY_5b9a27565c70 |= reg_nation__n_regionkey;
//Probe Hash table
HT_5b9a27565c70.for_each(KEY_5b9a27565c70, [&] __device__ (auto const SLOT_5b9a27565c70) {

auto const [slot_first5b9a27565c70, slot_second5b9a27565c70] = SLOT_5b9a27565c70;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5b9a27565ff0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5b9a27577ac0(uint64_t* BUF_5b9a27565c70, uint64_t* BUF_5b9a27565ff0, uint64_t* BUF_IDX_5b9a27565ff0, HASHTABLE_PROBE HT_5b9a27565c70, HASHTABLE_INSERT HT_5b9a27565ff0, DBI32Type* nation__n_nationkey, DBI32Type* nation__n_regionkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_5b9a27565c70 = 0;
auto reg_nation__n_regionkey = nation__n_regionkey[tid];

KEY_5b9a27565c70 |= reg_nation__n_regionkey;
//Probe Hash table
HT_5b9a27565c70.for_each(KEY_5b9a27565c70, [&] __device__ (auto const SLOT_5b9a27565c70) {
auto const [slot_first5b9a27565c70, slot_second5b9a27565c70] = SLOT_5b9a27565c70;
if (!(true)) return;
uint64_t KEY_5b9a27565ff0 = 0;
auto reg_nation__n_nationkey = nation__n_nationkey[tid];

KEY_5b9a27565ff0 |= reg_nation__n_nationkey;
// Insert hash table kernel;
auto buf_idx_5b9a27565ff0 = atomicAdd((int*)BUF_IDX_5b9a27565ff0, 1);
HT_5b9a27565ff0.insert(cuco::pair{KEY_5b9a27565ff0, buf_idx_5b9a27565ff0});
BUF_5b9a27565ff0[buf_idx_5b9a27565ff0 * 2 + 0] = BUF_5b9a27565c70[slot_second5b9a27565c70 * 1 + 0];
BUF_5b9a27565ff0[buf_idx_5b9a27565ff0 * 2 + 1] = tid;
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_5b9a27552fa0(uint64_t* BUF_5b9a27565ff0, uint64_t* COUNT5b9a2756eea0, HASHTABLE_PROBE HT_5b9a27565ff0, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_5b9a27565ff0 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_5b9a27565ff0 |= reg_customer__c_nationkey;
//Probe Hash table
HT_5b9a27565ff0.for_each(KEY_5b9a27565ff0, [&] __device__ (auto const SLOT_5b9a27565ff0) {

auto const [slot_first5b9a27565ff0, slot_second5b9a27565ff0] = SLOT_5b9a27565ff0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5b9a2756eea0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5b9a27552fa0(uint64_t* BUF_5b9a27565ff0, uint64_t* BUF_5b9a2756eea0, uint64_t* BUF_IDX_5b9a2756eea0, HASHTABLE_PROBE HT_5b9a27565ff0, HASHTABLE_INSERT HT_5b9a2756eea0, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_5b9a27565ff0 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_5b9a27565ff0 |= reg_customer__c_nationkey;
//Probe Hash table
HT_5b9a27565ff0.for_each(KEY_5b9a27565ff0, [&] __device__ (auto const SLOT_5b9a27565ff0) {
auto const [slot_first5b9a27565ff0, slot_second5b9a27565ff0] = SLOT_5b9a27565ff0;
if (!(true)) return;
uint64_t KEY_5b9a2756eea0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5b9a2756eea0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_5b9a2756eea0 = atomicAdd((int*)BUF_IDX_5b9a2756eea0, 1);
HT_5b9a2756eea0.insert(cuco::pair{KEY_5b9a2756eea0, buf_idx_5b9a2756eea0});
BUF_5b9a2756eea0[buf_idx_5b9a2756eea0 * 3 + 0] = tid;
BUF_5b9a2756eea0[buf_idx_5b9a2756eea0 * 3 + 1] = BUF_5b9a27565ff0[slot_second5b9a27565ff0 * 2 + 0];
BUF_5b9a2756eea0[buf_idx_5b9a2756eea0 * 3 + 2] = BUF_5b9a27565ff0[slot_second5b9a27565ff0 * 2 + 1];
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_5b9a27553510(uint64_t* BUF_5b9a2756eea0, uint64_t* COUNT5b9a2756efb0, HASHTABLE_PROBE HT_5b9a2756eea0, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 9131, Predicate::lt))) return;
uint64_t KEY_5b9a2756eea0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_5b9a2756eea0 |= reg_orders__o_custkey;
//Probe Hash table
HT_5b9a2756eea0.for_each(KEY_5b9a2756eea0, [&] __device__ (auto const SLOT_5b9a2756eea0) {

auto const [slot_first5b9a2756eea0, slot_second5b9a2756eea0] = SLOT_5b9a2756eea0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5b9a2756efb0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5b9a27553510(uint64_t* BUF_5b9a2756eea0, uint64_t* BUF_5b9a2756efb0, uint64_t* BUF_IDX_5b9a2756efb0, HASHTABLE_PROBE HT_5b9a2756eea0, HASHTABLE_INSERT HT_5b9a2756efb0, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 9131, Predicate::lt))) return;
uint64_t KEY_5b9a2756eea0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_5b9a2756eea0 |= reg_orders__o_custkey;
//Probe Hash table
HT_5b9a2756eea0.for_each(KEY_5b9a2756eea0, [&] __device__ (auto const SLOT_5b9a2756eea0) {
auto const [slot_first5b9a2756eea0, slot_second5b9a2756eea0] = SLOT_5b9a2756eea0;
if (!(true)) return;
uint64_t KEY_5b9a2756efb0 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_5b9a2756efb0 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_5b9a2756efb0 = atomicAdd((int*)BUF_IDX_5b9a2756efb0, 1);
HT_5b9a2756efb0.insert(cuco::pair{KEY_5b9a2756efb0, buf_idx_5b9a2756efb0});
BUF_5b9a2756efb0[buf_idx_5b9a2756efb0 * 4 + 0] = BUF_5b9a2756eea0[slot_second5b9a2756eea0 * 3 + 0];
BUF_5b9a2756efb0[buf_idx_5b9a2756efb0 * 4 + 1] = tid;
BUF_5b9a2756efb0[buf_idx_5b9a2756efb0 * 4 + 2] = BUF_5b9a2756eea0[slot_second5b9a2756eea0 * 3 + 1];
BUF_5b9a2756efb0[buf_idx_5b9a2756efb0 * 4 + 3] = BUF_5b9a2756eea0[slot_second5b9a2756eea0 * 3 + 2];
});
}
__global__ void count_5b9a27581e80(uint64_t* COUNT5b9a2756f0c0, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
//Materialize count
atomicAdd((int*)COUNT5b9a2756f0c0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5b9a27581e80(uint64_t* BUF_5b9a2756f0c0, uint64_t* BUF_IDX_5b9a2756f0c0, HASHTABLE_INSERT HT_5b9a2756f0c0, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_5b9a2756f0c0 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_5b9a2756f0c0 |= reg_supplier__s_nationkey;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];
KEY_5b9a2756f0c0 <<= 32;
KEY_5b9a2756f0c0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5b9a2756f0c0 = atomicAdd((int*)BUF_IDX_5b9a2756f0c0, 1);
HT_5b9a2756f0c0.insert(cuco::pair{KEY_5b9a2756f0c0, buf_idx_5b9a2756f0c0});
BUF_5b9a2756f0c0[buf_idx_5b9a2756f0c0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5b9a27574080(uint64_t* BUF_5b9a2756efb0, uint64_t* BUF_5b9a2756f0c0, HASHTABLE_INSERT HT_5b9a27521cc0, HASHTABLE_PROBE HT_5b9a2756efb0, HASHTABLE_PROBE HT_5b9a2756f0c0, DBI32Type* customer__c_nationkey, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_5b9a2756efb0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5b9a2756efb0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5b9a2756efb0.for_each(KEY_5b9a2756efb0, [&] __device__ (auto const SLOT_5b9a2756efb0) {

auto const [slot_first5b9a2756efb0, slot_second5b9a2756efb0] = SLOT_5b9a2756efb0;
if (!(true)) return;
uint64_t KEY_5b9a2756f0c0 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[BUF_5b9a2756efb0[slot_second5b9a2756efb0 * 4 + 0]];

KEY_5b9a2756f0c0 |= reg_customer__c_nationkey;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];
KEY_5b9a2756f0c0 <<= 32;
KEY_5b9a2756f0c0 |= reg_lineitem__l_suppkey;
//Probe Hash table
HT_5b9a2756f0c0.for_each(KEY_5b9a2756f0c0, [&] __device__ (auto const SLOT_5b9a2756f0c0) {

auto const [slot_first5b9a2756f0c0, slot_second5b9a2756f0c0] = SLOT_5b9a2756f0c0;
if (!(true)) return;
uint64_t KEY_5b9a27521cc0 = 0;
auto reg_nation__n_name_encoded = nation__n_name_encoded[BUF_5b9a2756efb0[slot_second5b9a2756efb0 * 4 + 3]];

KEY_5b9a27521cc0 |= reg_nation__n_name_encoded;
//Create aggregation hash table
HT_5b9a27521cc0.insert(cuco::pair{KEY_5b9a27521cc0, 1});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5b9a27574080(uint64_t* BUF_5b9a2756efb0, uint64_t* BUF_5b9a2756f0c0, HASHTABLE_FIND HT_5b9a27521cc0, HASHTABLE_PROBE HT_5b9a2756efb0, HASHTABLE_PROBE HT_5b9a2756f0c0, DBI16Type* KEY_5b9a27521cc0nation__n_name_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* customer__c_nationkey, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_5b9a2756efb0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5b9a2756efb0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5b9a2756efb0.for_each(KEY_5b9a2756efb0, [&] __device__ (auto const SLOT_5b9a2756efb0) {
auto const [slot_first5b9a2756efb0, slot_second5b9a2756efb0] = SLOT_5b9a2756efb0;
if (!(true)) return;
uint64_t KEY_5b9a2756f0c0 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[BUF_5b9a2756efb0[slot_second5b9a2756efb0 * 4 + 0]];

KEY_5b9a2756f0c0 |= reg_customer__c_nationkey;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];
KEY_5b9a2756f0c0 <<= 32;
KEY_5b9a2756f0c0 |= reg_lineitem__l_suppkey;
//Probe Hash table
HT_5b9a2756f0c0.for_each(KEY_5b9a2756f0c0, [&] __device__ (auto const SLOT_5b9a2756f0c0) {
auto const [slot_first5b9a2756f0c0, slot_second5b9a2756f0c0] = SLOT_5b9a2756f0c0;
if (!(true)) return;
uint64_t KEY_5b9a27521cc0 = 0;
auto reg_nation__n_name_encoded = nation__n_name_encoded[BUF_5b9a2756efb0[slot_second5b9a2756efb0 * 4 + 3]];

KEY_5b9a27521cc0 |= reg_nation__n_name_encoded;
//Aggregate in hashtable
auto buf_idx_5b9a27521cc0 = HT_5b9a27521cc0.find(KEY_5b9a27521cc0)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5b9a27521cc0], reg_map0__tmp_attr1);
KEY_5b9a27521cc0nation__n_name_encoded[buf_idx_5b9a27521cc0] = reg_nation__n_name_encoded;
});
});
}
__global__ void count_5b9a2758ce30(size_t COUNT5b9a27521cc0, uint64_t* COUNT5b9a27534e70) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5b9a27521cc0) return;
//Materialize count
atomicAdd((int*)COUNT5b9a27534e70, 1);
}
__global__ void main_5b9a2758ce30(size_t COUNT5b9a27521cc0, DBDecimalType* MAT5b9a27534e70aggr0__tmp_attr0, DBI16Type* MAT5b9a27534e70nation__n_name_encoded, uint64_t* MAT_IDX5b9a27534e70, DBDecimalType* aggr0__tmp_attr0, DBI16Type* nation__n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5b9a27521cc0) return;
//Materialize buffers
auto mat_idx5b9a27534e70 = atomicAdd((int*)MAT_IDX5b9a27534e70, 1);
auto reg_nation__n_name_encoded = nation__n_name_encoded[tid];
MAT5b9a27534e70nation__n_name_encoded[mat_idx5b9a27534e70] = reg_nation__n_name_encoded;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5b9a27534e70aggr0__tmp_attr0[mat_idx5b9a27534e70] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map) {
//Materialize count
uint64_t* d_COUNT5b9a27565c70;
hipMalloc(&d_COUNT5b9a27565c70, sizeof(uint64_t));
hipMemset(d_COUNT5b9a27565c70, 0, sizeof(uint64_t));
count_5b9a27576900<<<std::ceil((float)region_size/32.), 32>>>(d_COUNT5b9a27565c70, d_region__r_name, region_size);
uint64_t COUNT5b9a27565c70;
hipMemcpy(&COUNT5b9a27565c70, d_COUNT5b9a27565c70, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b9a27565c70;
hipMalloc(&d_BUF_IDX_5b9a27565c70, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b9a27565c70, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b9a27565c70;
hipMalloc(&d_BUF_5b9a27565c70, sizeof(uint64_t) * COUNT5b9a27565c70 * 1);
auto d_HT_5b9a27565c70 = cuco::experimental::static_multimap{ (int)COUNT5b9a27565c70*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b9a27576900<<<std::ceil((float)region_size/32.), 32>>>(d_BUF_5b9a27565c70, d_BUF_IDX_5b9a27565c70, d_HT_5b9a27565c70.ref(cuco::insert), d_region__r_name, d_region__r_regionkey, region_size);
//Materialize count
uint64_t* d_COUNT5b9a27565ff0;
hipMalloc(&d_COUNT5b9a27565ff0, sizeof(uint64_t));
hipMemset(d_COUNT5b9a27565ff0, 0, sizeof(uint64_t));
count_5b9a27577ac0<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_5b9a27565c70, d_COUNT5b9a27565ff0, d_HT_5b9a27565c70.ref(cuco::for_each), d_nation__n_regionkey, nation_size);
uint64_t COUNT5b9a27565ff0;
hipMemcpy(&COUNT5b9a27565ff0, d_COUNT5b9a27565ff0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b9a27565ff0;
hipMalloc(&d_BUF_IDX_5b9a27565ff0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b9a27565ff0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b9a27565ff0;
hipMalloc(&d_BUF_5b9a27565ff0, sizeof(uint64_t) * COUNT5b9a27565ff0 * 2);
auto d_HT_5b9a27565ff0 = cuco::experimental::static_multimap{ (int)COUNT5b9a27565ff0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b9a27577ac0<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_5b9a27565c70, d_BUF_5b9a27565ff0, d_BUF_IDX_5b9a27565ff0, d_HT_5b9a27565c70.ref(cuco::for_each), d_HT_5b9a27565ff0.ref(cuco::insert), d_nation__n_nationkey, d_nation__n_regionkey, nation_size);
//Materialize count
uint64_t* d_COUNT5b9a2756eea0;
hipMalloc(&d_COUNT5b9a2756eea0, sizeof(uint64_t));
hipMemset(d_COUNT5b9a2756eea0, 0, sizeof(uint64_t));
count_5b9a27552fa0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5b9a27565ff0, d_COUNT5b9a2756eea0, d_HT_5b9a27565ff0.ref(cuco::for_each), d_customer__c_nationkey, customer_size);
uint64_t COUNT5b9a2756eea0;
hipMemcpy(&COUNT5b9a2756eea0, d_COUNT5b9a2756eea0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b9a2756eea0;
hipMalloc(&d_BUF_IDX_5b9a2756eea0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b9a2756eea0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b9a2756eea0;
hipMalloc(&d_BUF_5b9a2756eea0, sizeof(uint64_t) * COUNT5b9a2756eea0 * 3);
auto d_HT_5b9a2756eea0 = cuco::experimental::static_multimap{ (int)COUNT5b9a2756eea0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b9a27552fa0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5b9a27565ff0, d_BUF_5b9a2756eea0, d_BUF_IDX_5b9a2756eea0, d_HT_5b9a27565ff0.ref(cuco::for_each), d_HT_5b9a2756eea0.ref(cuco::insert), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
//Materialize count
uint64_t* d_COUNT5b9a2756efb0;
hipMalloc(&d_COUNT5b9a2756efb0, sizeof(uint64_t));
hipMemset(d_COUNT5b9a2756efb0, 0, sizeof(uint64_t));
count_5b9a27553510<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5b9a2756eea0, d_COUNT5b9a2756efb0, d_HT_5b9a2756eea0.ref(cuco::for_each), d_orders__o_custkey, d_orders__o_orderdate, orders_size);
uint64_t COUNT5b9a2756efb0;
hipMemcpy(&COUNT5b9a2756efb0, d_COUNT5b9a2756efb0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b9a2756efb0;
hipMalloc(&d_BUF_IDX_5b9a2756efb0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b9a2756efb0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b9a2756efb0;
hipMalloc(&d_BUF_5b9a2756efb0, sizeof(uint64_t) * COUNT5b9a2756efb0 * 4);
auto d_HT_5b9a2756efb0 = cuco::experimental::static_multimap{ (int)COUNT5b9a2756efb0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b9a27553510<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5b9a2756eea0, d_BUF_5b9a2756efb0, d_BUF_IDX_5b9a2756efb0, d_HT_5b9a2756eea0.ref(cuco::for_each), d_HT_5b9a2756efb0.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
//Materialize count
uint64_t* d_COUNT5b9a2756f0c0;
hipMalloc(&d_COUNT5b9a2756f0c0, sizeof(uint64_t));
hipMemset(d_COUNT5b9a2756f0c0, 0, sizeof(uint64_t));
count_5b9a27581e80<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT5b9a2756f0c0, supplier_size);
uint64_t COUNT5b9a2756f0c0;
hipMemcpy(&COUNT5b9a2756f0c0, d_COUNT5b9a2756f0c0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b9a2756f0c0;
hipMalloc(&d_BUF_IDX_5b9a2756f0c0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b9a2756f0c0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b9a2756f0c0;
hipMalloc(&d_BUF_5b9a2756f0c0, sizeof(uint64_t) * COUNT5b9a2756f0c0 * 1);
auto d_HT_5b9a2756f0c0 = cuco::experimental::static_multimap{ (int)COUNT5b9a2756f0c0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b9a27581e80<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_5b9a2756f0c0, d_BUF_IDX_5b9a2756f0c0, d_HT_5b9a2756f0c0.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
//Create aggregation hash table
auto d_HT_5b9a27521cc0 = cuco::static_map{ (int)22857*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5b9a27574080<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5b9a2756efb0, d_BUF_5b9a2756f0c0, d_HT_5b9a27521cc0.ref(cuco::insert), d_HT_5b9a2756efb0.ref(cuco::for_each), d_HT_5b9a2756f0c0.ref(cuco::for_each), d_customer__c_nationkey, d_lineitem__l_orderkey, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded);
size_t COUNT5b9a27521cc0 = d_HT_5b9a27521cc0.size();
thrust::device_vector<int64_t> keys_5b9a27521cc0(COUNT5b9a27521cc0), vals_5b9a27521cc0(COUNT5b9a27521cc0);
d_HT_5b9a27521cc0.retrieve_all(keys_5b9a27521cc0.begin(), vals_5b9a27521cc0.begin());
d_HT_5b9a27521cc0.clear();
int64_t* raw_keys5b9a27521cc0 = thrust::raw_pointer_cast(keys_5b9a27521cc0.data());
insertKeys<<<std::ceil((float)COUNT5b9a27521cc0/32.), 32>>>(raw_keys5b9a27521cc0, d_HT_5b9a27521cc0.ref(cuco::insert), COUNT5b9a27521cc0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5b9a27521cc0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5b9a27521cc0);
DBI16Type* d_KEY_5b9a27521cc0nation__n_name_encoded;
hipMalloc(&d_KEY_5b9a27521cc0nation__n_name_encoded, sizeof(DBI16Type) * COUNT5b9a27521cc0);
hipMemset(d_KEY_5b9a27521cc0nation__n_name_encoded, 0, sizeof(DBI16Type) * COUNT5b9a27521cc0);
main_5b9a27574080<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5b9a2756efb0, d_BUF_5b9a2756f0c0, d_HT_5b9a27521cc0.ref(cuco::find), d_HT_5b9a2756efb0.ref(cuco::for_each), d_HT_5b9a2756f0c0.ref(cuco::for_each), d_KEY_5b9a27521cc0nation__n_name_encoded, d_aggr0__tmp_attr0, d_customer__c_nationkey, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded);
//Materialize count
uint64_t* d_COUNT5b9a27534e70;
hipMalloc(&d_COUNT5b9a27534e70, sizeof(uint64_t));
hipMemset(d_COUNT5b9a27534e70, 0, sizeof(uint64_t));
count_5b9a2758ce30<<<std::ceil((float)COUNT5b9a27521cc0/32.), 32>>>(COUNT5b9a27521cc0, d_COUNT5b9a27534e70);
uint64_t COUNT5b9a27534e70;
hipMemcpy(&COUNT5b9a27534e70, d_COUNT5b9a27534e70, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5b9a27534e70;
hipMalloc(&d_MAT_IDX5b9a27534e70, sizeof(uint64_t));
hipMemset(d_MAT_IDX5b9a27534e70, 0, sizeof(uint64_t));
auto MAT5b9a27534e70nation__n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5b9a27534e70);
DBI16Type* d_MAT5b9a27534e70nation__n_name_encoded;
hipMalloc(&d_MAT5b9a27534e70nation__n_name_encoded, sizeof(DBI16Type) * COUNT5b9a27534e70);
auto MAT5b9a27534e70aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5b9a27534e70);
DBDecimalType* d_MAT5b9a27534e70aggr0__tmp_attr0;
hipMalloc(&d_MAT5b9a27534e70aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5b9a27534e70);
main_5b9a2758ce30<<<std::ceil((float)COUNT5b9a27521cc0/32.), 32>>>(COUNT5b9a27521cc0, d_MAT5b9a27534e70aggr0__tmp_attr0, d_MAT5b9a27534e70nation__n_name_encoded, d_MAT_IDX5b9a27534e70, d_aggr0__tmp_attr0, d_KEY_5b9a27521cc0nation__n_name_encoded);
hipMemcpy(MAT5b9a27534e70nation__n_name_encoded, d_MAT5b9a27534e70nation__n_name_encoded, sizeof(DBI16Type) * COUNT5b9a27534e70, hipMemcpyDeviceToHost);
hipMemcpy(MAT5b9a27534e70aggr0__tmp_attr0, d_MAT5b9a27534e70aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5b9a27534e70, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5b9a27534e70; i++) { std::cout << nation__n_name_map[MAT5b9a27534e70nation__n_name_encoded[i]] << "\t";
std::cout << MAT5b9a27534e70aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_5b9a27565c70);
hipFree(d_BUF_IDX_5b9a27565c70);
hipFree(d_COUNT5b9a27565c70);
hipFree(d_BUF_5b9a27565ff0);
hipFree(d_BUF_IDX_5b9a27565ff0);
hipFree(d_COUNT5b9a27565ff0);
hipFree(d_BUF_5b9a2756eea0);
hipFree(d_BUF_IDX_5b9a2756eea0);
hipFree(d_COUNT5b9a2756eea0);
hipFree(d_BUF_5b9a2756efb0);
hipFree(d_BUF_IDX_5b9a2756efb0);
hipFree(d_COUNT5b9a2756efb0);
hipFree(d_BUF_5b9a2756f0c0);
hipFree(d_BUF_IDX_5b9a2756f0c0);
hipFree(d_COUNT5b9a2756f0c0);
hipFree(d_KEY_5b9a27521cc0nation__n_name_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5b9a27534e70);
hipFree(d_MAT5b9a27534e70aggr0__tmp_attr0);
hipFree(d_MAT5b9a27534e70nation__n_name_encoded);
hipFree(d_MAT_IDX5b9a27534e70);
free(MAT5b9a27534e70aggr0__tmp_attr0);
free(MAT5b9a27534e70nation__n_name_encoded);
}