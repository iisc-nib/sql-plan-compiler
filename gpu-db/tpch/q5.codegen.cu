#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_567196a4a0f0(uint64_t* COUNT567196a3a0b0, DBStringType* region__r_name, size_t region_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= region_size) return;
auto reg_region__r_name = region__r_name[tid];
if (!(evaluatePredicate(reg_region__r_name, "ASIA", Predicate::eq))) return;
//Materialize count
atomicAdd((int*)COUNT567196a3a0b0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_567196a4a0f0(uint64_t* BUF_567196a3a0b0, uint64_t* BUF_IDX_567196a3a0b0, HASHTABLE_INSERT HT_567196a3a0b0, DBStringType* region__r_name, DBI32Type* region__r_regionkey, size_t region_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= region_size) return;
auto reg_region__r_name = region__r_name[tid];
if (!(evaluatePredicate(reg_region__r_name, "ASIA", Predicate::eq))) return;
uint64_t KEY_567196a3a0b0 = 0;
auto reg_region__r_regionkey = region__r_regionkey[tid];

KEY_567196a3a0b0 |= reg_region__r_regionkey;
// Insert hash table kernel;
auto buf_idx_567196a3a0b0 = atomicAdd((int*)BUF_IDX_567196a3a0b0, 1);
HT_567196a3a0b0.insert(cuco::pair{KEY_567196a3a0b0, buf_idx_567196a3a0b0});
BUF_567196a3a0b0[buf_idx_567196a3a0b0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_567196a4b450(uint64_t* BUF_567196a3a0b0, uint64_t* COUNT567196a39af0, HASHTABLE_PROBE HT_567196a3a0b0, DBI32Type* nation__n_regionkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_567196a3a0b0 = 0;
auto reg_nation__n_regionkey = nation__n_regionkey[tid];

KEY_567196a3a0b0 |= reg_nation__n_regionkey;
//Probe Hash table
HT_567196a3a0b0.for_each(KEY_567196a3a0b0, [&] __device__ (auto const SLOT_567196a3a0b0) {

auto const [slot_first567196a3a0b0, slot_second567196a3a0b0] = SLOT_567196a3a0b0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT567196a39af0, 1);
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_567196a4b450(uint64_t* BUF_567196a39af0, uint64_t* BUF_567196a3a0b0, uint64_t* BUF_IDX_567196a39af0, HASHTABLE_INSERT HT_567196a39af0, HASHTABLE_PROBE HT_567196a3a0b0, DBI32Type* nation__n_nationkey, DBI32Type* nation__n_regionkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_567196a3a0b0 = 0;
auto reg_nation__n_regionkey = nation__n_regionkey[tid];

KEY_567196a3a0b0 |= reg_nation__n_regionkey;
//Probe Hash table
HT_567196a3a0b0.for_each(KEY_567196a3a0b0, [&] __device__ (auto const SLOT_567196a3a0b0) {
auto const [slot_first567196a3a0b0, slot_second567196a3a0b0] = SLOT_567196a3a0b0;
if (!(true)) return;
uint64_t KEY_567196a39af0 = 0;
auto reg_nation__n_nationkey = nation__n_nationkey[tid];

KEY_567196a39af0 |= reg_nation__n_nationkey;
// Insert hash table kernel;
auto buf_idx_567196a39af0 = atomicAdd((int*)BUF_IDX_567196a39af0, 1);
HT_567196a39af0.insert(cuco::pair{KEY_567196a39af0, buf_idx_567196a39af0});
BUF_567196a39af0[buf_idx_567196a39af0 * 2 + 0] = BUF_567196a3a0b0[slot_second567196a3a0b0 * 1 + 0];
BUF_567196a39af0[buf_idx_567196a39af0 * 2 + 1] = tid;
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_567196a26fd0(uint64_t* BUF_567196a39af0, uint64_t* COUNT567196a422c0, HASHTABLE_PROBE HT_567196a39af0, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_567196a39af0 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_567196a39af0 |= reg_customer__c_nationkey;
//Probe Hash table
HT_567196a39af0.for_each(KEY_567196a39af0, [&] __device__ (auto const SLOT_567196a39af0) {

auto const [slot_first567196a39af0, slot_second567196a39af0] = SLOT_567196a39af0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT567196a422c0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_567196a26fd0(uint64_t* BUF_567196a39af0, uint64_t* BUF_567196a422c0, uint64_t* BUF_IDX_567196a422c0, HASHTABLE_PROBE HT_567196a39af0, HASHTABLE_INSERT HT_567196a422c0, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_567196a39af0 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_567196a39af0 |= reg_customer__c_nationkey;
//Probe Hash table
HT_567196a39af0.for_each(KEY_567196a39af0, [&] __device__ (auto const SLOT_567196a39af0) {
auto const [slot_first567196a39af0, slot_second567196a39af0] = SLOT_567196a39af0;
if (!(true)) return;
uint64_t KEY_567196a422c0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_567196a422c0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_567196a422c0 = atomicAdd((int*)BUF_IDX_567196a422c0, 1);
HT_567196a422c0.insert(cuco::pair{KEY_567196a422c0, buf_idx_567196a422c0});
BUF_567196a422c0[buf_idx_567196a422c0 * 3 + 0] = tid;
BUF_567196a422c0[buf_idx_567196a422c0 * 3 + 1] = BUF_567196a39af0[slot_second567196a39af0 * 2 + 0];
BUF_567196a422c0[buf_idx_567196a422c0 * 3 + 2] = BUF_567196a39af0[slot_second567196a39af0 * 2 + 1];
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_567196a27540(uint64_t* BUF_567196a422c0, uint64_t* COUNT567196a42430, HASHTABLE_PROBE HT_567196a422c0, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 9131, Predicate::lt))) return;
uint64_t KEY_567196a422c0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_567196a422c0 |= reg_orders__o_custkey;
//Probe Hash table
HT_567196a422c0.for_each(KEY_567196a422c0, [&] __device__ (auto const SLOT_567196a422c0) {

auto const [slot_first567196a422c0, slot_second567196a422c0] = SLOT_567196a422c0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT567196a42430, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_567196a27540(uint64_t* BUF_567196a422c0, uint64_t* BUF_567196a42430, uint64_t* BUF_IDX_567196a42430, HASHTABLE_PROBE HT_567196a422c0, HASHTABLE_INSERT HT_567196a42430, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 9131, Predicate::lt))) return;
uint64_t KEY_567196a422c0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_567196a422c0 |= reg_orders__o_custkey;
//Probe Hash table
HT_567196a422c0.for_each(KEY_567196a422c0, [&] __device__ (auto const SLOT_567196a422c0) {
auto const [slot_first567196a422c0, slot_second567196a422c0] = SLOT_567196a422c0;
if (!(true)) return;
uint64_t KEY_567196a42430 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_567196a42430 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_567196a42430 = atomicAdd((int*)BUF_IDX_567196a42430, 1);
HT_567196a42430.insert(cuco::pair{KEY_567196a42430, buf_idx_567196a42430});
BUF_567196a42430[buf_idx_567196a42430 * 4 + 0] = BUF_567196a422c0[slot_second567196a422c0 * 3 + 0];
BUF_567196a42430[buf_idx_567196a42430 * 4 + 1] = tid;
BUF_567196a42430[buf_idx_567196a42430 * 4 + 2] = BUF_567196a422c0[slot_second567196a422c0 * 3 + 1];
BUF_567196a42430[buf_idx_567196a42430 * 4 + 3] = BUF_567196a422c0[slot_second567196a422c0 * 3 + 2];
});
}
__global__ void count_567196a559b0(uint64_t* COUNT567196a42540, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
//Materialize count
atomicAdd((int*)COUNT567196a42540, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_567196a559b0(uint64_t* BUF_567196a42540, uint64_t* BUF_IDX_567196a42540, HASHTABLE_INSERT HT_567196a42540, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_567196a42540 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_567196a42540 |= reg_supplier__s_suppkey;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];
KEY_567196a42540 <<= 32;
KEY_567196a42540 |= reg_supplier__s_nationkey;
// Insert hash table kernel;
auto buf_idx_567196a42540 = atomicAdd((int*)BUF_IDX_567196a42540, 1);
HT_567196a42540.insert(cuco::pair{KEY_567196a42540, buf_idx_567196a42540});
BUF_567196a42540[buf_idx_567196a42540 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_567196a473f0(uint64_t* BUF_567196a42430, uint64_t* BUF_567196a42540, HASHTABLE_INSERT HT_5671969f5da0, HASHTABLE_PROBE HT_567196a42430, HASHTABLE_PROBE HT_567196a42540, DBI32Type* customer__c_nationkey, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_567196a42430 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_567196a42430 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_567196a42430.for_each(KEY_567196a42430, [&] __device__ (auto const SLOT_567196a42430) {

auto const [slot_first567196a42430, slot_second567196a42430] = SLOT_567196a42430;
if (!(true)) return;
uint64_t KEY_567196a42540 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_567196a42540 |= reg_lineitem__l_suppkey;
auto reg_customer__c_nationkey = customer__c_nationkey[BUF_567196a42430[slot_second567196a42430 * 4 + 0]];
KEY_567196a42540 <<= 32;
KEY_567196a42540 |= reg_customer__c_nationkey;
//Probe Hash table
HT_567196a42540.for_each(KEY_567196a42540, [&] __device__ (auto const SLOT_567196a42540) {

auto const [slot_first567196a42540, slot_second567196a42540] = SLOT_567196a42540;
if (!(true)) return;
uint64_t KEY_5671969f5da0 = 0;
auto reg_nation__n_name_encoded = nation__n_name_encoded[BUF_567196a42430[slot_second567196a42430 * 4 + 3]];

KEY_5671969f5da0 |= reg_nation__n_name_encoded;
//Create aggregation hash table
HT_5671969f5da0.insert(cuco::pair{KEY_5671969f5da0, 1});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_567196a473f0(uint64_t* BUF_567196a42430, uint64_t* BUF_567196a42540, HASHTABLE_FIND HT_5671969f5da0, HASHTABLE_PROBE HT_567196a42430, HASHTABLE_PROBE HT_567196a42540, DBI16Type* KEY_5671969f5da0nation__n_name_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* customer__c_nationkey, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_567196a42430 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_567196a42430 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_567196a42430.for_each(KEY_567196a42430, [&] __device__ (auto const SLOT_567196a42430) {
auto const [slot_first567196a42430, slot_second567196a42430] = SLOT_567196a42430;
if (!(true)) return;
uint64_t KEY_567196a42540 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_567196a42540 |= reg_lineitem__l_suppkey;
auto reg_customer__c_nationkey = customer__c_nationkey[BUF_567196a42430[slot_second567196a42430 * 4 + 0]];
KEY_567196a42540 <<= 32;
KEY_567196a42540 |= reg_customer__c_nationkey;
//Probe Hash table
HT_567196a42540.for_each(KEY_567196a42540, [&] __device__ (auto const SLOT_567196a42540) {
auto const [slot_first567196a42540, slot_second567196a42540] = SLOT_567196a42540;
if (!(true)) return;
uint64_t KEY_5671969f5da0 = 0;
auto reg_nation__n_name_encoded = nation__n_name_encoded[BUF_567196a42430[slot_second567196a42430 * 4 + 3]];

KEY_5671969f5da0 |= reg_nation__n_name_encoded;
//Aggregate in hashtable
auto buf_idx_5671969f5da0 = HT_5671969f5da0.find(KEY_5671969f5da0)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5671969f5da0], reg_map0__tmp_attr1);
KEY_5671969f5da0nation__n_name_encoded[buf_idx_5671969f5da0] = reg_nation__n_name_encoded;
});
});
}
__global__ void count_567196a60360(size_t COUNT5671969f5da0, uint64_t* COUNT567196a09930) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5671969f5da0) return;
//Materialize count
atomicAdd((int*)COUNT567196a09930, 1);
}
__global__ void main_567196a60360(size_t COUNT5671969f5da0, DBDecimalType* MAT567196a09930aggr0__tmp_attr0, DBI16Type* MAT567196a09930nation__n_name_encoded, uint64_t* MAT_IDX567196a09930, DBDecimalType* aggr0__tmp_attr0, DBI16Type* nation__n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5671969f5da0) return;
//Materialize buffers
auto mat_idx567196a09930 = atomicAdd((int*)MAT_IDX567196a09930, 1);
auto reg_nation__n_name_encoded = nation__n_name_encoded[tid];
MAT567196a09930nation__n_name_encoded[mat_idx567196a09930] = reg_nation__n_name_encoded;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT567196a09930aggr0__tmp_attr0[mat_idx567196a09930] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map) {
//Materialize count
uint64_t* d_COUNT567196a3a0b0;
hipMalloc(&d_COUNT567196a3a0b0, sizeof(uint64_t));
hipMemset(d_COUNT567196a3a0b0, 0, sizeof(uint64_t));
count_567196a4a0f0<<<std::ceil((float)region_size/32.), 32>>>(d_COUNT567196a3a0b0, d_region__r_name, region_size);
uint64_t COUNT567196a3a0b0;
hipMemcpy(&COUNT567196a3a0b0, d_COUNT567196a3a0b0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_567196a3a0b0;
hipMalloc(&d_BUF_IDX_567196a3a0b0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_567196a3a0b0, 0, sizeof(uint64_t));
uint64_t* d_BUF_567196a3a0b0;
hipMalloc(&d_BUF_567196a3a0b0, sizeof(uint64_t) * COUNT567196a3a0b0 * 1);
auto d_HT_567196a3a0b0 = cuco::experimental::static_multimap{ (int)COUNT567196a3a0b0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_567196a4a0f0<<<std::ceil((float)region_size/32.), 32>>>(d_BUF_567196a3a0b0, d_BUF_IDX_567196a3a0b0, d_HT_567196a3a0b0.ref(cuco::insert), d_region__r_name, d_region__r_regionkey, region_size);
//Materialize count
uint64_t* d_COUNT567196a39af0;
hipMalloc(&d_COUNT567196a39af0, sizeof(uint64_t));
hipMemset(d_COUNT567196a39af0, 0, sizeof(uint64_t));
count_567196a4b450<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_567196a3a0b0, d_COUNT567196a39af0, d_HT_567196a3a0b0.ref(cuco::for_each), d_nation__n_regionkey, nation_size);
uint64_t COUNT567196a39af0;
hipMemcpy(&COUNT567196a39af0, d_COUNT567196a39af0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_567196a39af0;
hipMalloc(&d_BUF_IDX_567196a39af0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_567196a39af0, 0, sizeof(uint64_t));
uint64_t* d_BUF_567196a39af0;
hipMalloc(&d_BUF_567196a39af0, sizeof(uint64_t) * COUNT567196a39af0 * 2);
auto d_HT_567196a39af0 = cuco::experimental::static_multimap{ (int)COUNT567196a39af0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_567196a4b450<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_567196a39af0, d_BUF_567196a3a0b0, d_BUF_IDX_567196a39af0, d_HT_567196a39af0.ref(cuco::insert), d_HT_567196a3a0b0.ref(cuco::for_each), d_nation__n_nationkey, d_nation__n_regionkey, nation_size);
//Materialize count
uint64_t* d_COUNT567196a422c0;
hipMalloc(&d_COUNT567196a422c0, sizeof(uint64_t));
hipMemset(d_COUNT567196a422c0, 0, sizeof(uint64_t));
count_567196a26fd0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_567196a39af0, d_COUNT567196a422c0, d_HT_567196a39af0.ref(cuco::for_each), d_customer__c_nationkey, customer_size);
uint64_t COUNT567196a422c0;
hipMemcpy(&COUNT567196a422c0, d_COUNT567196a422c0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_567196a422c0;
hipMalloc(&d_BUF_IDX_567196a422c0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_567196a422c0, 0, sizeof(uint64_t));
uint64_t* d_BUF_567196a422c0;
hipMalloc(&d_BUF_567196a422c0, sizeof(uint64_t) * COUNT567196a422c0 * 3);
auto d_HT_567196a422c0 = cuco::experimental::static_multimap{ (int)COUNT567196a422c0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_567196a26fd0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_567196a39af0, d_BUF_567196a422c0, d_BUF_IDX_567196a422c0, d_HT_567196a39af0.ref(cuco::for_each), d_HT_567196a422c0.ref(cuco::insert), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
//Materialize count
uint64_t* d_COUNT567196a42430;
hipMalloc(&d_COUNT567196a42430, sizeof(uint64_t));
hipMemset(d_COUNT567196a42430, 0, sizeof(uint64_t));
count_567196a27540<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_567196a422c0, d_COUNT567196a42430, d_HT_567196a422c0.ref(cuco::for_each), d_orders__o_custkey, d_orders__o_orderdate, orders_size);
uint64_t COUNT567196a42430;
hipMemcpy(&COUNT567196a42430, d_COUNT567196a42430, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_567196a42430;
hipMalloc(&d_BUF_IDX_567196a42430, sizeof(uint64_t));
hipMemset(d_BUF_IDX_567196a42430, 0, sizeof(uint64_t));
uint64_t* d_BUF_567196a42430;
hipMalloc(&d_BUF_567196a42430, sizeof(uint64_t) * COUNT567196a42430 * 4);
auto d_HT_567196a42430 = cuco::experimental::static_multimap{ (int)COUNT567196a42430*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_567196a27540<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_567196a422c0, d_BUF_567196a42430, d_BUF_IDX_567196a42430, d_HT_567196a422c0.ref(cuco::for_each), d_HT_567196a42430.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
//Materialize count
uint64_t* d_COUNT567196a42540;
hipMalloc(&d_COUNT567196a42540, sizeof(uint64_t));
hipMemset(d_COUNT567196a42540, 0, sizeof(uint64_t));
count_567196a559b0<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT567196a42540, supplier_size);
uint64_t COUNT567196a42540;
hipMemcpy(&COUNT567196a42540, d_COUNT567196a42540, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_567196a42540;
hipMalloc(&d_BUF_IDX_567196a42540, sizeof(uint64_t));
hipMemset(d_BUF_IDX_567196a42540, 0, sizeof(uint64_t));
uint64_t* d_BUF_567196a42540;
hipMalloc(&d_BUF_567196a42540, sizeof(uint64_t) * COUNT567196a42540 * 1);
auto d_HT_567196a42540 = cuco::experimental::static_multimap{ (int)COUNT567196a42540*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_567196a559b0<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_567196a42540, d_BUF_IDX_567196a42540, d_HT_567196a42540.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
//Create aggregation hash table
auto d_HT_5671969f5da0 = cuco::static_map{ (int)22857*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_567196a473f0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_567196a42430, d_BUF_567196a42540, d_HT_5671969f5da0.ref(cuco::insert), d_HT_567196a42430.ref(cuco::for_each), d_HT_567196a42540.ref(cuco::for_each), d_customer__c_nationkey, d_lineitem__l_orderkey, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded);
size_t COUNT5671969f5da0 = d_HT_5671969f5da0.size();
thrust::device_vector<int64_t> keys_5671969f5da0(COUNT5671969f5da0), vals_5671969f5da0(COUNT5671969f5da0);
d_HT_5671969f5da0.retrieve_all(keys_5671969f5da0.begin(), vals_5671969f5da0.begin());
d_HT_5671969f5da0.clear();
int64_t* raw_keys5671969f5da0 = thrust::raw_pointer_cast(keys_5671969f5da0.data());
insertKeys<<<std::ceil((float)COUNT5671969f5da0/32.), 32>>>(raw_keys5671969f5da0, d_HT_5671969f5da0.ref(cuco::insert), COUNT5671969f5da0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5671969f5da0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5671969f5da0);
DBI16Type* d_KEY_5671969f5da0nation__n_name_encoded;
hipMalloc(&d_KEY_5671969f5da0nation__n_name_encoded, sizeof(DBI16Type) * COUNT5671969f5da0);
hipMemset(d_KEY_5671969f5da0nation__n_name_encoded, 0, sizeof(DBI16Type) * COUNT5671969f5da0);
main_567196a473f0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_567196a42430, d_BUF_567196a42540, d_HT_5671969f5da0.ref(cuco::find), d_HT_567196a42430.ref(cuco::for_each), d_HT_567196a42540.ref(cuco::for_each), d_KEY_5671969f5da0nation__n_name_encoded, d_aggr0__tmp_attr0, d_customer__c_nationkey, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded);
//Materialize count
uint64_t* d_COUNT567196a09930;
hipMalloc(&d_COUNT567196a09930, sizeof(uint64_t));
hipMemset(d_COUNT567196a09930, 0, sizeof(uint64_t));
count_567196a60360<<<std::ceil((float)COUNT5671969f5da0/32.), 32>>>(COUNT5671969f5da0, d_COUNT567196a09930);
uint64_t COUNT567196a09930;
hipMemcpy(&COUNT567196a09930, d_COUNT567196a09930, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX567196a09930;
hipMalloc(&d_MAT_IDX567196a09930, sizeof(uint64_t));
hipMemset(d_MAT_IDX567196a09930, 0, sizeof(uint64_t));
auto MAT567196a09930nation__n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT567196a09930);
DBI16Type* d_MAT567196a09930nation__n_name_encoded;
hipMalloc(&d_MAT567196a09930nation__n_name_encoded, sizeof(DBI16Type) * COUNT567196a09930);
auto MAT567196a09930aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT567196a09930);
DBDecimalType* d_MAT567196a09930aggr0__tmp_attr0;
hipMalloc(&d_MAT567196a09930aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT567196a09930);
main_567196a60360<<<std::ceil((float)COUNT5671969f5da0/32.), 32>>>(COUNT5671969f5da0, d_MAT567196a09930aggr0__tmp_attr0, d_MAT567196a09930nation__n_name_encoded, d_MAT_IDX567196a09930, d_aggr0__tmp_attr0, d_KEY_5671969f5da0nation__n_name_encoded);
hipMemcpy(MAT567196a09930nation__n_name_encoded, d_MAT567196a09930nation__n_name_encoded, sizeof(DBI16Type) * COUNT567196a09930, hipMemcpyDeviceToHost);
hipMemcpy(MAT567196a09930aggr0__tmp_attr0, d_MAT567196a09930aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT567196a09930, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT567196a09930; i++) { std::cout << nation__n_name_map[MAT567196a09930nation__n_name_encoded[i]] << "\t";
std::cout << MAT567196a09930aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_567196a3a0b0);
hipFree(d_BUF_IDX_567196a3a0b0);
hipFree(d_COUNT567196a3a0b0);
hipFree(d_BUF_567196a39af0);
hipFree(d_BUF_IDX_567196a39af0);
hipFree(d_COUNT567196a39af0);
hipFree(d_BUF_567196a422c0);
hipFree(d_BUF_IDX_567196a422c0);
hipFree(d_COUNT567196a422c0);
hipFree(d_BUF_567196a42430);
hipFree(d_BUF_IDX_567196a42430);
hipFree(d_COUNT567196a42430);
hipFree(d_BUF_567196a42540);
hipFree(d_BUF_IDX_567196a42540);
hipFree(d_COUNT567196a42540);
hipFree(d_KEY_5671969f5da0nation__n_name_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT567196a09930);
hipFree(d_MAT567196a09930aggr0__tmp_attr0);
hipFree(d_MAT567196a09930nation__n_name_encoded);
hipFree(d_MAT_IDX567196a09930);
free(MAT567196a09930aggr0__tmp_attr0);
free(MAT567196a09930nation__n_name_encoded);
}