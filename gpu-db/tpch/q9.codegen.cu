#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_59055ebe58d0(uint64_t* COUNT59055ebd7580, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
//Materialize count
atomicAdd((int*)COUNT59055ebd7580, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_59055ebe58d0(uint64_t* BUF_59055ebd7580, uint64_t* BUF_IDX_59055ebd7580, HASHTABLE_INSERT HT_59055ebd7580, DBI32Type* nation__n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_59055ebd7580 = 0;
auto reg_nation__n_nationkey = nation__n_nationkey[tid];

KEY_59055ebd7580 |= reg_nation__n_nationkey;
// Insert hash table kernel;
auto buf_idx_59055ebd7580 = atomicAdd((int*)BUF_IDX_59055ebd7580, 1);
HT_59055ebd7580.insert(cuco::pair{KEY_59055ebd7580, buf_idx_59055ebd7580});
BUF_59055ebd7580[buf_idx_59055ebd7580 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_59055ebc2740(uint64_t* BUF_59055ebd7580, uint64_t* COUNT59055ebd63c0, HASHTABLE_PROBE HT_59055ebd7580, DBI32Type* supplier__s_nationkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_59055ebd7580 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_59055ebd7580 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_59055ebd7580.for_each(KEY_59055ebd7580, [&] __device__ (auto const SLOT_59055ebd7580) {

auto const [slot_first59055ebd7580, slot_second59055ebd7580] = SLOT_59055ebd7580;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT59055ebd63c0, 1);
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_59055ebc2740(uint64_t* BUF_59055ebd63c0, uint64_t* BUF_59055ebd7580, uint64_t* BUF_IDX_59055ebd63c0, HASHTABLE_INSERT HT_59055ebd63c0, HASHTABLE_PROBE HT_59055ebd7580, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_59055ebd7580 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_59055ebd7580 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_59055ebd7580.for_each(KEY_59055ebd7580, [&] __device__ (auto const SLOT_59055ebd7580) {
auto const [slot_first59055ebd7580, slot_second59055ebd7580] = SLOT_59055ebd7580;
if (!(true)) return;
uint64_t KEY_59055ebd63c0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_59055ebd63c0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_59055ebd63c0 = atomicAdd((int*)BUF_IDX_59055ebd63c0, 1);
HT_59055ebd63c0.insert(cuco::pair{KEY_59055ebd63c0, buf_idx_59055ebd63c0});
BUF_59055ebd63c0[buf_idx_59055ebd63c0 * 2 + 0] = tid;
BUF_59055ebd63c0[buf_idx_59055ebd63c0 * 2 + 1] = BUF_59055ebd7580[slot_second59055ebd7580 * 1 + 0];
});
}
__global__ void count_59055ebec730(uint64_t* COUNT59055ebdd8d0, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
//Materialize count
atomicAdd((int*)COUNT59055ebdd8d0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_59055ebec730(uint64_t* BUF_59055ebdd8d0, uint64_t* BUF_IDX_59055ebdd8d0, HASHTABLE_INSERT HT_59055ebdd8d0, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_59055ebdd8d0 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_59055ebdd8d0 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_59055ebdd8d0 = atomicAdd((int*)BUF_IDX_59055ebdd8d0, 1);
HT_59055ebdd8d0.insert(cuco::pair{KEY_59055ebdd8d0, buf_idx_59055ebdd8d0});
BUF_59055ebdd8d0[buf_idx_59055ebdd8d0 * 1 + 0] = tid;
}
__global__ void count_59055ebc2110(uint64_t* COUNT59055ebdd990, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
//Materialize count
atomicAdd((int*)COUNT59055ebdd990, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_59055ebc2110(uint64_t* BUF_59055ebdd990, uint64_t* BUF_IDX_59055ebdd990, HASHTABLE_INSERT HT_59055ebdd990, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
uint64_t KEY_59055ebdd990 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_59055ebdd990 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_59055ebdd990 = atomicAdd((int*)BUF_IDX_59055ebdd990, 1);
HT_59055ebdd990.insert(cuco::pair{KEY_59055ebdd990, buf_idx_59055ebdd990});
BUF_59055ebdd990[buf_idx_59055ebdd990 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_59055ebe4890(uint64_t* BUF_59055ebd63c0, uint64_t* COUNT59055ebddaa0, HASHTABLE_PROBE HT_59055ebd63c0, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= partsupp_size) return;
uint64_t KEY_59055ebd63c0 = 0;
auto reg_partsupp__ps_suppkey = partsupp__ps_suppkey[tid];

KEY_59055ebd63c0 |= reg_partsupp__ps_suppkey;
//Probe Hash table
HT_59055ebd63c0.for_each(KEY_59055ebd63c0, [&] __device__ (auto const SLOT_59055ebd63c0) {

auto const [slot_first59055ebd63c0, slot_second59055ebd63c0] = SLOT_59055ebd63c0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT59055ebddaa0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_59055ebe4890(uint64_t* BUF_59055ebd63c0, uint64_t* BUF_59055ebddaa0, uint64_t* BUF_IDX_59055ebddaa0, HASHTABLE_PROBE HT_59055ebd63c0, HASHTABLE_INSERT HT_59055ebddaa0, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size, DBI32Type* supplier__s_suppkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= partsupp_size) return;
uint64_t KEY_59055ebd63c0 = 0;
auto reg_partsupp__ps_suppkey = partsupp__ps_suppkey[tid];

KEY_59055ebd63c0 |= reg_partsupp__ps_suppkey;
//Probe Hash table
HT_59055ebd63c0.for_each(KEY_59055ebd63c0, [&] __device__ (auto const SLOT_59055ebd63c0) {
auto const [slot_first59055ebd63c0, slot_second59055ebd63c0] = SLOT_59055ebd63c0;
if (!(true)) return;
uint64_t KEY_59055ebddaa0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[BUF_59055ebd63c0[slot_second59055ebd63c0 * 2 + 0]];

KEY_59055ebddaa0 |= reg_supplier__s_suppkey;
auto reg_partsupp__ps_partkey = partsupp__ps_partkey[tid];
KEY_59055ebddaa0 <<= 32;
KEY_59055ebddaa0 |= reg_partsupp__ps_partkey;
// Insert hash table kernel;
auto buf_idx_59055ebddaa0 = atomicAdd((int*)BUF_IDX_59055ebddaa0, 1);
HT_59055ebddaa0.insert(cuco::pair{KEY_59055ebddaa0, buf_idx_59055ebddaa0});
BUF_59055ebddaa0[buf_idx_59055ebddaa0 * 3 + 0] = BUF_59055ebd63c0[slot_second59055ebd63c0 * 2 + 0];
BUF_59055ebddaa0[buf_idx_59055ebddaa0 * 3 + 1] = tid;
BUF_59055ebddaa0[buf_idx_59055ebddaa0 * 3 + 2] = BUF_59055ebd63c0[slot_second59055ebd63c0 * 2 + 1];
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_59055ebe2030(uint64_t* BUF_59055ebdd8d0, uint64_t* BUF_59055ebdd990, uint64_t* BUF_59055ebddaa0, HASHTABLE_INSERT HT_59055eb91bb0, HASHTABLE_PROBE HT_59055ebdd8d0, HASHTABLE_PROBE HT_59055ebdd990, HASHTABLE_PROBE HT_59055ebddaa0, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded, DBDateType* orders__o_orderdate) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_59055ebdd8d0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_59055ebdd8d0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_59055ebdd8d0.for_each(KEY_59055ebdd8d0, [&] __device__ (auto const SLOT_59055ebdd8d0) {

auto const [slot_first59055ebdd8d0, slot_second59055ebdd8d0] = SLOT_59055ebdd8d0;
if (!(true)) return;
uint64_t KEY_59055ebdd990 = 0;
auto reg_lineitem__l_partkey = lineitem__l_partkey[tid];

KEY_59055ebdd990 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_59055ebdd990.for_each(KEY_59055ebdd990, [&] __device__ (auto const SLOT_59055ebdd990) {

auto const [slot_first59055ebdd990, slot_second59055ebdd990] = SLOT_59055ebdd990;
if (!(true)) return;
uint64_t KEY_59055ebddaa0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_59055ebddaa0 |= reg_lineitem__l_suppkey;
KEY_59055ebddaa0 <<= 32;
KEY_59055ebddaa0 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_59055ebddaa0.for_each(KEY_59055ebddaa0, [&] __device__ (auto const SLOT_59055ebddaa0) {

auto const [slot_first59055ebddaa0, slot_second59055ebddaa0] = SLOT_59055ebddaa0;
if (!(true)) return;
uint64_t KEY_59055eb91bb0 = 0;
auto reg_nation__n_name_encoded = nation__n_name_encoded[BUF_59055ebddaa0[slot_second59055ebddaa0 * 3 + 2]];

KEY_59055eb91bb0 |= reg_nation__n_name_encoded;
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_59055ebdd8d0[slot_second59055ebdd8d0 * 1 + 0]];
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_orders__o_orderdate);
KEY_59055eb91bb0 <<= 32;
KEY_59055eb91bb0 |= (DBI32Type)reg_map0__tmp_attr0;
//Create aggregation hash table
HT_59055eb91bb0.insert(cuco::pair{KEY_59055eb91bb0, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_59055ebe2030(uint64_t* BUF_59055ebdd8d0, uint64_t* BUF_59055ebdd990, uint64_t* BUF_59055ebddaa0, HASHTABLE_FIND HT_59055eb91bb0, HASHTABLE_PROBE HT_59055ebdd8d0, HASHTABLE_PROBE HT_59055ebdd990, HASHTABLE_PROBE HT_59055ebddaa0, DBI64Type* KEY_59055eb91bb0map0__tmp_attr0, DBI16Type* KEY_59055eb91bb0nation__n_name_encoded, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded, DBDateType* orders__o_orderdate, DBDecimalType* partsupp__ps_supplycost) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_59055ebdd8d0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_59055ebdd8d0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_59055ebdd8d0.for_each(KEY_59055ebdd8d0, [&] __device__ (auto const SLOT_59055ebdd8d0) {
auto const [slot_first59055ebdd8d0, slot_second59055ebdd8d0] = SLOT_59055ebdd8d0;
if (!(true)) return;
uint64_t KEY_59055ebdd990 = 0;
auto reg_lineitem__l_partkey = lineitem__l_partkey[tid];

KEY_59055ebdd990 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_59055ebdd990.for_each(KEY_59055ebdd990, [&] __device__ (auto const SLOT_59055ebdd990) {
auto const [slot_first59055ebdd990, slot_second59055ebdd990] = SLOT_59055ebdd990;
if (!(true)) return;
uint64_t KEY_59055ebddaa0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_59055ebddaa0 |= reg_lineitem__l_suppkey;
KEY_59055ebddaa0 <<= 32;
KEY_59055ebddaa0 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_59055ebddaa0.for_each(KEY_59055ebddaa0, [&] __device__ (auto const SLOT_59055ebddaa0) {
auto const [slot_first59055ebddaa0, slot_second59055ebddaa0] = SLOT_59055ebddaa0;
if (!(true)) return;
uint64_t KEY_59055eb91bb0 = 0;
auto reg_nation__n_name_encoded = nation__n_name_encoded[BUF_59055ebddaa0[slot_second59055ebddaa0 * 3 + 2]];

KEY_59055eb91bb0 |= reg_nation__n_name_encoded;
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_59055ebdd8d0[slot_second59055ebdd8d0 * 1 + 0]];
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_orders__o_orderdate);
KEY_59055eb91bb0 <<= 32;
KEY_59055eb91bb0 |= (DBI32Type)reg_map0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_59055eb91bb0 = HT_59055eb91bb0.find(KEY_59055eb91bb0)->second;
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
auto reg_partsupp__ps_supplycost = partsupp__ps_supplycost[BUF_59055ebddaa0[slot_second59055ebddaa0 * 3 + 1]];
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = ((reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount))) - ((reg_partsupp__ps_supplycost) * (reg_lineitem__l_quantity));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_59055eb91bb0], reg_map0__tmp_attr1);
KEY_59055eb91bb0nation__n_name_encoded[buf_idx_59055eb91bb0] = reg_nation__n_name_encoded;
KEY_59055eb91bb0map0__tmp_attr0[buf_idx_59055eb91bb0] = reg_map0__tmp_attr0;
});
});
});
}
__global__ void count_59055ebfb750(size_t COUNT59055eb91bb0, uint64_t* COUNT59055eba5290) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT59055eb91bb0) return;
//Materialize count
atomicAdd((int*)COUNT59055eba5290, 1);
}
__global__ void main_59055ebfb750(size_t COUNT59055eb91bb0, DBDecimalType* MAT59055eba5290aggr0__tmp_attr2, DBI64Type* MAT59055eba5290map0__tmp_attr0, DBI16Type* MAT59055eba5290nation__n_name_encoded, uint64_t* MAT_IDX59055eba5290, DBDecimalType* aggr0__tmp_attr2, DBI64Type* map0__tmp_attr0, DBI16Type* nation__n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT59055eb91bb0) return;
//Materialize buffers
auto mat_idx59055eba5290 = atomicAdd((int*)MAT_IDX59055eba5290, 1);
auto reg_nation__n_name_encoded = nation__n_name_encoded[tid];
MAT59055eba5290nation__n_name_encoded[mat_idx59055eba5290] = reg_nation__n_name_encoded;
auto reg_map0__tmp_attr0 = map0__tmp_attr0[tid];
MAT59055eba5290map0__tmp_attr0[mat_idx59055eba5290] = reg_map0__tmp_attr0;
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT59055eba5290aggr0__tmp_attr2[mat_idx59055eba5290] = reg_aggr0__tmp_attr2;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map) {
//Materialize count
uint64_t* d_COUNT59055ebd7580;
hipMalloc(&d_COUNT59055ebd7580, sizeof(uint64_t));
hipMemset(d_COUNT59055ebd7580, 0, sizeof(uint64_t));
count_59055ebe58d0<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT59055ebd7580, nation_size);
uint64_t COUNT59055ebd7580;
hipMemcpy(&COUNT59055ebd7580, d_COUNT59055ebd7580, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_59055ebd7580;
hipMalloc(&d_BUF_IDX_59055ebd7580, sizeof(uint64_t));
hipMemset(d_BUF_IDX_59055ebd7580, 0, sizeof(uint64_t));
uint64_t* d_BUF_59055ebd7580;
hipMalloc(&d_BUF_59055ebd7580, sizeof(uint64_t) * COUNT59055ebd7580 * 1);
auto d_HT_59055ebd7580 = cuco::experimental::static_multimap{ (int)COUNT59055ebd7580*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_59055ebe58d0<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_59055ebd7580, d_BUF_IDX_59055ebd7580, d_HT_59055ebd7580.ref(cuco::insert), d_nation__n_nationkey, nation_size);
//Materialize count
uint64_t* d_COUNT59055ebd63c0;
hipMalloc(&d_COUNT59055ebd63c0, sizeof(uint64_t));
hipMemset(d_COUNT59055ebd63c0, 0, sizeof(uint64_t));
count_59055ebc2740<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_59055ebd7580, d_COUNT59055ebd63c0, d_HT_59055ebd7580.ref(cuco::for_each), d_supplier__s_nationkey, supplier_size);
uint64_t COUNT59055ebd63c0;
hipMemcpy(&COUNT59055ebd63c0, d_COUNT59055ebd63c0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_59055ebd63c0;
hipMalloc(&d_BUF_IDX_59055ebd63c0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_59055ebd63c0, 0, sizeof(uint64_t));
uint64_t* d_BUF_59055ebd63c0;
hipMalloc(&d_BUF_59055ebd63c0, sizeof(uint64_t) * COUNT59055ebd63c0 * 2);
auto d_HT_59055ebd63c0 = cuco::experimental::static_multimap{ (int)COUNT59055ebd63c0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_59055ebc2740<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_59055ebd63c0, d_BUF_59055ebd7580, d_BUF_IDX_59055ebd63c0, d_HT_59055ebd63c0.ref(cuco::insert), d_HT_59055ebd7580.ref(cuco::for_each), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT59055ebdd8d0;
hipMalloc(&d_COUNT59055ebdd8d0, sizeof(uint64_t));
hipMemset(d_COUNT59055ebdd8d0, 0, sizeof(uint64_t));
count_59055ebec730<<<std::ceil((float)orders_size/32.), 32>>>(d_COUNT59055ebdd8d0, orders_size);
uint64_t COUNT59055ebdd8d0;
hipMemcpy(&COUNT59055ebdd8d0, d_COUNT59055ebdd8d0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_59055ebdd8d0;
hipMalloc(&d_BUF_IDX_59055ebdd8d0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_59055ebdd8d0, 0, sizeof(uint64_t));
uint64_t* d_BUF_59055ebdd8d0;
hipMalloc(&d_BUF_59055ebdd8d0, sizeof(uint64_t) * COUNT59055ebdd8d0 * 1);
auto d_HT_59055ebdd8d0 = cuco::experimental::static_multimap{ (int)COUNT59055ebdd8d0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_59055ebec730<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_59055ebdd8d0, d_BUF_IDX_59055ebdd8d0, d_HT_59055ebdd8d0.ref(cuco::insert), d_orders__o_orderkey, orders_size);
//Materialize count
uint64_t* d_COUNT59055ebdd990;
hipMalloc(&d_COUNT59055ebdd990, sizeof(uint64_t));
hipMemset(d_COUNT59055ebdd990, 0, sizeof(uint64_t));
count_59055ebc2110<<<std::ceil((float)part_size/32.), 32>>>(d_COUNT59055ebdd990, part_size);
uint64_t COUNT59055ebdd990;
hipMemcpy(&COUNT59055ebdd990, d_COUNT59055ebdd990, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_59055ebdd990;
hipMalloc(&d_BUF_IDX_59055ebdd990, sizeof(uint64_t));
hipMemset(d_BUF_IDX_59055ebdd990, 0, sizeof(uint64_t));
uint64_t* d_BUF_59055ebdd990;
hipMalloc(&d_BUF_59055ebdd990, sizeof(uint64_t) * COUNT59055ebdd990 * 1);
auto d_HT_59055ebdd990 = cuco::experimental::static_multimap{ (int)COUNT59055ebdd990*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_59055ebc2110<<<std::ceil((float)part_size/32.), 32>>>(d_BUF_59055ebdd990, d_BUF_IDX_59055ebdd990, d_HT_59055ebdd990.ref(cuco::insert), d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT59055ebddaa0;
hipMalloc(&d_COUNT59055ebddaa0, sizeof(uint64_t));
hipMemset(d_COUNT59055ebddaa0, 0, sizeof(uint64_t));
count_59055ebe4890<<<std::ceil((float)partsupp_size/32.), 32>>>(d_BUF_59055ebd63c0, d_COUNT59055ebddaa0, d_HT_59055ebd63c0.ref(cuco::for_each), d_partsupp__ps_suppkey, partsupp_size);
uint64_t COUNT59055ebddaa0;
hipMemcpy(&COUNT59055ebddaa0, d_COUNT59055ebddaa0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_59055ebddaa0;
hipMalloc(&d_BUF_IDX_59055ebddaa0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_59055ebddaa0, 0, sizeof(uint64_t));
uint64_t* d_BUF_59055ebddaa0;
hipMalloc(&d_BUF_59055ebddaa0, sizeof(uint64_t) * COUNT59055ebddaa0 * 3);
auto d_HT_59055ebddaa0 = cuco::experimental::static_multimap{ (int)COUNT59055ebddaa0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_59055ebe4890<<<std::ceil((float)partsupp_size/32.), 32>>>(d_BUF_59055ebd63c0, d_BUF_59055ebddaa0, d_BUF_IDX_59055ebddaa0, d_HT_59055ebd63c0.ref(cuco::for_each), d_HT_59055ebddaa0.ref(cuco::insert), d_partsupp__ps_partkey, d_partsupp__ps_suppkey, partsupp_size, d_supplier__s_suppkey);
//Create aggregation hash table
auto d_HT_59055eb91bb0 = cuco::static_map{ (int)48009721*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_59055ebe2030<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_59055ebdd8d0, d_BUF_59055ebdd990, d_BUF_59055ebddaa0, d_HT_59055eb91bb0.ref(cuco::insert), d_HT_59055ebdd8d0.ref(cuco::for_each), d_HT_59055ebdd990.ref(cuco::for_each), d_HT_59055ebddaa0.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded, d_orders__o_orderdate);
size_t COUNT59055eb91bb0 = d_HT_59055eb91bb0.size();
thrust::device_vector<int64_t> keys_59055eb91bb0(COUNT59055eb91bb0), vals_59055eb91bb0(COUNT59055eb91bb0);
d_HT_59055eb91bb0.retrieve_all(keys_59055eb91bb0.begin(), vals_59055eb91bb0.begin());
d_HT_59055eb91bb0.clear();
int64_t* raw_keys59055eb91bb0 = thrust::raw_pointer_cast(keys_59055eb91bb0.data());
insertKeys<<<std::ceil((float)COUNT59055eb91bb0/32.), 32>>>(raw_keys59055eb91bb0, d_HT_59055eb91bb0.ref(cuco::insert), COUNT59055eb91bb0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT59055eb91bb0);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT59055eb91bb0);
DBI16Type* d_KEY_59055eb91bb0nation__n_name_encoded;
hipMalloc(&d_KEY_59055eb91bb0nation__n_name_encoded, sizeof(DBI16Type) * COUNT59055eb91bb0);
hipMemset(d_KEY_59055eb91bb0nation__n_name_encoded, 0, sizeof(DBI16Type) * COUNT59055eb91bb0);
DBI64Type* d_KEY_59055eb91bb0map0__tmp_attr0;
hipMalloc(&d_KEY_59055eb91bb0map0__tmp_attr0, sizeof(DBI64Type) * COUNT59055eb91bb0);
hipMemset(d_KEY_59055eb91bb0map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT59055eb91bb0);
main_59055ebe2030<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_59055ebdd8d0, d_BUF_59055ebdd990, d_BUF_59055ebddaa0, d_HT_59055eb91bb0.ref(cuco::find), d_HT_59055ebdd8d0.ref(cuco::for_each), d_HT_59055ebdd990.ref(cuco::for_each), d_HT_59055ebddaa0.ref(cuco::for_each), d_KEY_59055eb91bb0map0__tmp_attr0, d_KEY_59055eb91bb0nation__n_name_encoded, d_aggr0__tmp_attr2, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_quantity, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded, d_orders__o_orderdate, d_partsupp__ps_supplycost);
//Materialize count
uint64_t* d_COUNT59055eba5290;
hipMalloc(&d_COUNT59055eba5290, sizeof(uint64_t));
hipMemset(d_COUNT59055eba5290, 0, sizeof(uint64_t));
count_59055ebfb750<<<std::ceil((float)COUNT59055eb91bb0/32.), 32>>>(COUNT59055eb91bb0, d_COUNT59055eba5290);
uint64_t COUNT59055eba5290;
hipMemcpy(&COUNT59055eba5290, d_COUNT59055eba5290, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX59055eba5290;
hipMalloc(&d_MAT_IDX59055eba5290, sizeof(uint64_t));
hipMemset(d_MAT_IDX59055eba5290, 0, sizeof(uint64_t));
auto MAT59055eba5290nation__n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT59055eba5290);
DBI16Type* d_MAT59055eba5290nation__n_name_encoded;
hipMalloc(&d_MAT59055eba5290nation__n_name_encoded, sizeof(DBI16Type) * COUNT59055eba5290);
auto MAT59055eba5290map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT59055eba5290);
DBI64Type* d_MAT59055eba5290map0__tmp_attr0;
hipMalloc(&d_MAT59055eba5290map0__tmp_attr0, sizeof(DBI64Type) * COUNT59055eba5290);
auto MAT59055eba5290aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT59055eba5290);
DBDecimalType* d_MAT59055eba5290aggr0__tmp_attr2;
hipMalloc(&d_MAT59055eba5290aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT59055eba5290);
main_59055ebfb750<<<std::ceil((float)COUNT59055eb91bb0/32.), 32>>>(COUNT59055eb91bb0, d_MAT59055eba5290aggr0__tmp_attr2, d_MAT59055eba5290map0__tmp_attr0, d_MAT59055eba5290nation__n_name_encoded, d_MAT_IDX59055eba5290, d_aggr0__tmp_attr2, d_KEY_59055eb91bb0map0__tmp_attr0, d_KEY_59055eb91bb0nation__n_name_encoded);
hipMemcpy(MAT59055eba5290nation__n_name_encoded, d_MAT59055eba5290nation__n_name_encoded, sizeof(DBI16Type) * COUNT59055eba5290, hipMemcpyDeviceToHost);
hipMemcpy(MAT59055eba5290map0__tmp_attr0, d_MAT59055eba5290map0__tmp_attr0, sizeof(DBI64Type) * COUNT59055eba5290, hipMemcpyDeviceToHost);
hipMemcpy(MAT59055eba5290aggr0__tmp_attr2, d_MAT59055eba5290aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT59055eba5290, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT59055eba5290; i++) { std::cout << nation__n_name_map[MAT59055eba5290nation__n_name_encoded[i]] << "\t";
std::cout << MAT59055eba5290map0__tmp_attr0[i] << "\t";
std::cout << MAT59055eba5290aggr0__tmp_attr2[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_59055ebd7580);
hipFree(d_BUF_IDX_59055ebd7580);
hipFree(d_COUNT59055ebd7580);
hipFree(d_BUF_59055ebd63c0);
hipFree(d_BUF_IDX_59055ebd63c0);
hipFree(d_COUNT59055ebd63c0);
hipFree(d_BUF_59055ebdd8d0);
hipFree(d_BUF_IDX_59055ebdd8d0);
hipFree(d_COUNT59055ebdd8d0);
hipFree(d_BUF_59055ebdd990);
hipFree(d_BUF_IDX_59055ebdd990);
hipFree(d_COUNT59055ebdd990);
hipFree(d_BUF_59055ebddaa0);
hipFree(d_BUF_IDX_59055ebddaa0);
hipFree(d_COUNT59055ebddaa0);
hipFree(d_KEY_59055eb91bb0map0__tmp_attr0);
hipFree(d_KEY_59055eb91bb0nation__n_name_encoded);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_COUNT59055eba5290);
hipFree(d_MAT59055eba5290aggr0__tmp_attr2);
hipFree(d_MAT59055eba5290map0__tmp_attr0);
hipFree(d_MAT59055eba5290nation__n_name_encoded);
hipFree(d_MAT_IDX59055eba5290);
free(MAT59055eba5290aggr0__tmp_attr2);
free(MAT59055eba5290map0__tmp_attr0);
free(MAT59055eba5290nation__n_name_encoded);
}