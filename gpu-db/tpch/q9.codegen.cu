#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_1(uint64_t* COUNT0, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
//Materialize count
atomicAdd((int*)COUNT0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBI32Type* nation__n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_0 = 0;
auto reg_nation__n_nationkey = nation__n_nationkey[tid];

KEY_0 |= reg_nation__n_nationkey;
// Insert hash table kernel;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0, buf_idx_0});
BUF_0[buf_idx_0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_3(uint64_t* BUF_0, uint64_t* COUNT2, HASHTABLE_PROBE HT_0, DBI32Type* supplier__s_nationkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_0 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_0 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_0.for_each(KEY_0, [&] __device__ (auto const SLOT_0) {

auto const [slot_first0, slot_second0] = SLOT_0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT2, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_0, uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_PROBE HT_0, HASHTABLE_INSERT HT_2, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_0 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_0 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_0.for_each(KEY_0, [&] __device__ (auto const SLOT_0) {
auto const [slot_first0, slot_second0] = SLOT_0;
if (!(true)) return;
uint64_t KEY_2 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_2 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_2 = atomicAdd((int*)BUF_IDX_2, 1);
HT_2.insert(cuco::pair{KEY_2, buf_idx_2});
BUF_2[buf_idx_2 * 2 + 0] = tid;
BUF_2[buf_idx_2 * 2 + 1] = BUF_0[slot_second0 * 1 + 0];
});
}
__global__ void count_5(uint64_t* COUNT4, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
//Materialize count
atomicAdd((int*)COUNT4, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_INSERT HT_4, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_4 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_4 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_4 = atomicAdd((int*)BUF_IDX_4, 1);
HT_4.insert(cuco::pair{KEY_4, buf_idx_4});
BUF_4[buf_idx_4 * 1 + 0] = tid;
}
__global__ void count_7(uint64_t* COUNT6, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
//Materialize count
atomicAdd((int*)COUNT6, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_7(uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_INSERT HT_6, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
uint64_t KEY_6 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_6 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_6 = atomicAdd((int*)BUF_IDX_6, 1);
HT_6.insert(cuco::pair{KEY_6, buf_idx_6});
BUF_6[buf_idx_6 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_9(uint64_t* BUF_2, uint64_t* COUNT8, HASHTABLE_PROBE HT_2, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= partsupp_size) return;
uint64_t KEY_2 = 0;
auto reg_partsupp__ps_suppkey = partsupp__ps_suppkey[tid];

KEY_2 |= reg_partsupp__ps_suppkey;
//Probe Hash table
HT_2.for_each(KEY_2, [&] __device__ (auto const SLOT_2) {

auto const [slot_first2, slot_second2] = SLOT_2;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT8, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_9(uint64_t* BUF_2, uint64_t* BUF_8, uint64_t* BUF_IDX_8, HASHTABLE_PROBE HT_2, HASHTABLE_INSERT HT_8, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size, DBI32Type* supplier__s_suppkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= partsupp_size) return;
uint64_t KEY_2 = 0;
auto reg_partsupp__ps_suppkey = partsupp__ps_suppkey[tid];

KEY_2 |= reg_partsupp__ps_suppkey;
//Probe Hash table
HT_2.for_each(KEY_2, [&] __device__ (auto const SLOT_2) {
auto const [slot_first2, slot_second2] = SLOT_2;
if (!(true)) return;
uint64_t KEY_8 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[BUF_2[slot_second2 * 2 + 0]];

KEY_8 |= reg_supplier__s_suppkey;
auto reg_partsupp__ps_partkey = partsupp__ps_partkey[tid];
KEY_8 <<= 32;
KEY_8 |= reg_partsupp__ps_partkey;
// Insert hash table kernel;
auto buf_idx_8 = atomicAdd((int*)BUF_IDX_8, 1);
HT_8.insert(cuco::pair{KEY_8, buf_idx_8});
BUF_8[buf_idx_8 * 3 + 0] = BUF_2[slot_second2 * 2 + 0];
BUF_8[buf_idx_8 * 3 + 1] = tid;
BUF_8[buf_idx_8 * 3 + 2] = BUF_2[slot_second2 * 2 + 1];
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_11(uint64_t* BUF_4, uint64_t* BUF_6, uint64_t* BUF_8, HASHTABLE_INSERT HT_10, HASHTABLE_PROBE HT_4, HASHTABLE_PROBE HT_6, HASHTABLE_PROBE HT_8, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded, DBDateType* orders__o_orderdate) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_4 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_4 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_4.for_each(KEY_4, [&] __device__ (auto const SLOT_4) {

auto const [slot_first4, slot_second4] = SLOT_4;
if (!(true)) return;
uint64_t KEY_6 = 0;
auto reg_lineitem__l_partkey = lineitem__l_partkey[tid];

KEY_6 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_6.for_each(KEY_6, [&] __device__ (auto const SLOT_6) {

auto const [slot_first6, slot_second6] = SLOT_6;
if (!(true)) return;
uint64_t KEY_8 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_8 |= reg_lineitem__l_suppkey;
KEY_8 <<= 32;
KEY_8 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_8.for_each(KEY_8, [&] __device__ (auto const SLOT_8) {

auto const [slot_first8, slot_second8] = SLOT_8;
if (!(true)) return;
uint64_t KEY_10 = 0;
auto reg_nation__n_name_encoded = nation__n_name_encoded[BUF_8[slot_second8 * 3 + 2]];

KEY_10 |= reg_nation__n_name_encoded;
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_4[slot_second4 * 1 + 0]];
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_orders__o_orderdate);
KEY_10 <<= 32;
KEY_10 |= (DBI32Type)reg_map0__tmp_attr0;
//Create aggregation hash table
HT_10.insert(cuco::pair{KEY_10, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_11(uint64_t* BUF_4, uint64_t* BUF_6, uint64_t* BUF_8, HASHTABLE_FIND HT_10, HASHTABLE_PROBE HT_4, HASHTABLE_PROBE HT_6, HASHTABLE_PROBE HT_8, DBI64Type* KEY_10map0__tmp_attr0, DBI16Type* KEY_10nation__n_name_encoded, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded, DBDateType* orders__o_orderdate, DBDecimalType* partsupp__ps_supplycost) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_4 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_4 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_4.for_each(KEY_4, [&] __device__ (auto const SLOT_4) {
auto const [slot_first4, slot_second4] = SLOT_4;
if (!(true)) return;
uint64_t KEY_6 = 0;
auto reg_lineitem__l_partkey = lineitem__l_partkey[tid];

KEY_6 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_6.for_each(KEY_6, [&] __device__ (auto const SLOT_6) {
auto const [slot_first6, slot_second6] = SLOT_6;
if (!(true)) return;
uint64_t KEY_8 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_8 |= reg_lineitem__l_suppkey;
KEY_8 <<= 32;
KEY_8 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_8.for_each(KEY_8, [&] __device__ (auto const SLOT_8) {
auto const [slot_first8, slot_second8] = SLOT_8;
if (!(true)) return;
uint64_t KEY_10 = 0;
auto reg_nation__n_name_encoded = nation__n_name_encoded[BUF_8[slot_second8 * 3 + 2]];

KEY_10 |= reg_nation__n_name_encoded;
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_4[slot_second4 * 1 + 0]];
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_orders__o_orderdate);
KEY_10 <<= 32;
KEY_10 |= (DBI32Type)reg_map0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_10 = HT_10.find(KEY_10)->second;
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
auto reg_partsupp__ps_supplycost = partsupp__ps_supplycost[BUF_8[slot_second8 * 3 + 1]];
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = ((reg_lineitem__l_extendedprice) * ((1.0) - (reg_lineitem__l_discount))) - ((reg_partsupp__ps_supplycost) * (reg_lineitem__l_quantity));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_10], reg_map0__tmp_attr1);
KEY_10nation__n_name_encoded[buf_idx_10] = reg_nation__n_name_encoded;
KEY_10map0__tmp_attr0[buf_idx_10] = reg_map0__tmp_attr0;
});
});
});
}
__global__ void count_13(size_t COUNT10, uint64_t* COUNT12) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT10) return;
//Materialize count
atomicAdd((int*)COUNT12, 1);
}
__global__ void main_13(size_t COUNT10, DBDecimalType* MAT12aggr0__tmp_attr2, DBI64Type* MAT12map0__tmp_attr0, DBI16Type* MAT12nation__n_name_encoded, uint64_t* MAT_IDX12, DBDecimalType* aggr0__tmp_attr2, DBI64Type* map0__tmp_attr0, DBI16Type* nation__n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT10) return;
//Materialize buffers
auto mat_idx12 = atomicAdd((int*)MAT_IDX12, 1);
auto reg_nation__n_name_encoded = nation__n_name_encoded[tid];
MAT12nation__n_name_encoded[mat_idx12] = reg_nation__n_name_encoded;
auto reg_map0__tmp_attr0 = map0__tmp_attr0[tid];
MAT12map0__tmp_attr0[mat_idx12] = reg_map0__tmp_attr0;
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT12aggr0__tmp_attr2[mat_idx12] = reg_aggr0__tmp_attr2;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)nation_size/128.), 128>>>(d_COUNT0, nation_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::experimental::static_multimap{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)nation_size/128.), 128>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_nation__n_nationkey, nation_size);
//Materialize count
uint64_t* d_COUNT2;
hipMalloc(&d_COUNT2, sizeof(uint64_t));
hipMemset(d_COUNT2, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_0, d_COUNT2, d_HT_0.ref(cuco::for_each), d_supplier__s_nationkey, supplier_size);
uint64_t COUNT2;
hipMemcpy(&COUNT2, d_COUNT2, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 2);
auto d_HT_2 = cuco::experimental::static_multimap{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_0, d_BUF_2, d_BUF_IDX_2, d_HT_0.ref(cuco::for_each), d_HT_2.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT4;
hipMalloc(&d_COUNT4, sizeof(uint64_t));
hipMemset(d_COUNT4, 0, sizeof(uint64_t));
count_5<<<std::ceil((float)orders_size/128.), 128>>>(d_COUNT4, orders_size);
uint64_t COUNT4;
hipMemcpy(&COUNT4, d_COUNT4, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 1);
auto d_HT_4 = cuco::experimental::static_multimap{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)orders_size/128.), 128>>>(d_BUF_4, d_BUF_IDX_4, d_HT_4.ref(cuco::insert), d_orders__o_orderkey, orders_size);
//Materialize count
uint64_t* d_COUNT6;
hipMalloc(&d_COUNT6, sizeof(uint64_t));
hipMemset(d_COUNT6, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)part_size/128.), 128>>>(d_COUNT6, part_size);
uint64_t COUNT6;
hipMemcpy(&COUNT6, d_COUNT6, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 1);
auto d_HT_6 = cuco::experimental::static_multimap{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_6, d_BUF_IDX_6, d_HT_6.ref(cuco::insert), d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT8;
hipMalloc(&d_COUNT8, sizeof(uint64_t));
hipMemset(d_COUNT8, 0, sizeof(uint64_t));
count_9<<<std::ceil((float)partsupp_size/128.), 128>>>(d_BUF_2, d_COUNT8, d_HT_2.ref(cuco::for_each), d_partsupp__ps_suppkey, partsupp_size);
uint64_t COUNT8;
hipMemcpy(&COUNT8, d_COUNT8, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_8;
hipMalloc(&d_BUF_IDX_8, sizeof(uint64_t));
hipMemset(d_BUF_IDX_8, 0, sizeof(uint64_t));
uint64_t* d_BUF_8;
hipMalloc(&d_BUF_8, sizeof(uint64_t) * COUNT8 * 3);
auto d_HT_8 = cuco::experimental::static_multimap{ (int)COUNT8*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_9<<<std::ceil((float)partsupp_size/128.), 128>>>(d_BUF_2, d_BUF_8, d_BUF_IDX_8, d_HT_2.ref(cuco::for_each), d_HT_8.ref(cuco::insert), d_partsupp__ps_partkey, d_partsupp__ps_suppkey, partsupp_size, d_supplier__s_suppkey);
//Create aggregation hash table
auto d_HT_10 = cuco::static_map{ (int)48009721*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_11<<<std::ceil((float)lineitem_size/128.), 128>>>(d_BUF_4, d_BUF_6, d_BUF_8, d_HT_10.ref(cuco::insert), d_HT_4.ref(cuco::for_each), d_HT_6.ref(cuco::for_each), d_HT_8.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded, d_orders__o_orderdate);
size_t COUNT10 = d_HT_10.size();
thrust::device_vector<int64_t> keys_10(COUNT10), vals_10(COUNT10);
d_HT_10.retrieve_all(keys_10.begin(), vals_10.begin());
d_HT_10.clear();
int64_t* raw_keys10 = thrust::raw_pointer_cast(keys_10.data());
insertKeys<<<std::ceil((float)COUNT10/128.), 128>>>(raw_keys10, d_HT_10.ref(cuco::insert), COUNT10);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT10);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT10);
DBI16Type* d_KEY_10nation__n_name_encoded;
hipMalloc(&d_KEY_10nation__n_name_encoded, sizeof(DBI16Type) * COUNT10);
hipMemset(d_KEY_10nation__n_name_encoded, 0, sizeof(DBI16Type) * COUNT10);
DBI64Type* d_KEY_10map0__tmp_attr0;
hipMalloc(&d_KEY_10map0__tmp_attr0, sizeof(DBI64Type) * COUNT10);
hipMemset(d_KEY_10map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT10);
main_11<<<std::ceil((float)lineitem_size/128.), 128>>>(d_BUF_4, d_BUF_6, d_BUF_8, d_HT_10.ref(cuco::find), d_HT_4.ref(cuco::for_each), d_HT_6.ref(cuco::for_each), d_HT_8.ref(cuco::for_each), d_KEY_10map0__tmp_attr0, d_KEY_10nation__n_name_encoded, d_aggr0__tmp_attr2, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_quantity, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded, d_orders__o_orderdate, d_partsupp__ps_supplycost);
//Materialize count
uint64_t* d_COUNT12;
hipMalloc(&d_COUNT12, sizeof(uint64_t));
hipMemset(d_COUNT12, 0, sizeof(uint64_t));
count_13<<<std::ceil((float)COUNT10/128.), 128>>>(COUNT10, d_COUNT12);
uint64_t COUNT12;
hipMemcpy(&COUNT12, d_COUNT12, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX12;
hipMalloc(&d_MAT_IDX12, sizeof(uint64_t));
hipMemset(d_MAT_IDX12, 0, sizeof(uint64_t));
auto MAT12nation__n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT12);
DBI16Type* d_MAT12nation__n_name_encoded;
hipMalloc(&d_MAT12nation__n_name_encoded, sizeof(DBI16Type) * COUNT12);
auto MAT12map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT12);
DBI64Type* d_MAT12map0__tmp_attr0;
hipMalloc(&d_MAT12map0__tmp_attr0, sizeof(DBI64Type) * COUNT12);
auto MAT12aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT12);
DBDecimalType* d_MAT12aggr0__tmp_attr2;
hipMalloc(&d_MAT12aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT12);
main_13<<<std::ceil((float)COUNT10/128.), 128>>>(COUNT10, d_MAT12aggr0__tmp_attr2, d_MAT12map0__tmp_attr0, d_MAT12nation__n_name_encoded, d_MAT_IDX12, d_aggr0__tmp_attr2, d_KEY_10map0__tmp_attr0, d_KEY_10nation__n_name_encoded);
hipMemcpy(MAT12nation__n_name_encoded, d_MAT12nation__n_name_encoded, sizeof(DBI16Type) * COUNT12, hipMemcpyDeviceToHost);
hipMemcpy(MAT12map0__tmp_attr0, d_MAT12map0__tmp_attr0, sizeof(DBI64Type) * COUNT12, hipMemcpyDeviceToHost);
hipMemcpy(MAT12aggr0__tmp_attr2, d_MAT12aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT12, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT12; i++) { std::cout << "" << nation__n_name_map[MAT12nation__n_name_encoded[i]];
std::cout << "|" << MAT12map0__tmp_attr0[i];
std::cout << "|" << MAT12aggr0__tmp_attr2[i];
std::cout << std::endl; }
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_COUNT0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_COUNT2);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_COUNT4);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_COUNT6);
hipFree(d_BUF_8);
hipFree(d_BUF_IDX_8);
hipFree(d_COUNT8);
hipFree(d_KEY_10map0__tmp_attr0);
hipFree(d_KEY_10nation__n_name_encoded);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_COUNT12);
hipFree(d_MAT12aggr0__tmp_attr2);
hipFree(d_MAT12map0__tmp_attr0);
hipFree(d_MAT12nation__n_name_encoded);
hipFree(d_MAT_IDX12);
free(MAT12aggr0__tmp_attr2);
free(MAT12map0__tmp_attr0);
free(MAT12nation__n_name_encoded);
}