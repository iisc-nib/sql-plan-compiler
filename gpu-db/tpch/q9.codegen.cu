#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5f1f02980ad0(uint64_t* COUNT5f1f0296f570, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
//Materialize count
atomicAdd((int*)COUNT5f1f0296f570, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5f1f02980ad0(uint64_t* BUF_5f1f0296f570, uint64_t* BUF_IDX_5f1f0296f570, HASHTABLE_INSERT HT_5f1f0296f570, DBI32Type* nation__n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_5f1f0296f570 = 0;
auto reg_nation__n_nationkey = nation__n_nationkey[tid];

KEY_5f1f0296f570 |= reg_nation__n_nationkey;
// Insert hash table kernel;
auto buf_idx_5f1f0296f570 = atomicAdd((int*)BUF_IDX_5f1f0296f570, 1);
HT_5f1f0296f570.insert(cuco::pair{KEY_5f1f0296f570, buf_idx_5f1f0296f570});
BUF_5f1f0296f570[buf_idx_5f1f0296f570 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5f1f0295b9f0(uint64_t* BUF_5f1f0296f570, uint64_t* COUNT5f1f0296f690, HASHTABLE_PROBE HT_5f1f0296f570, DBI32Type* supplier__s_nationkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_5f1f0296f570 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_5f1f0296f570 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_5f1f0296f570.for_each(KEY_5f1f0296f570, [&] __device__ (auto const SLOT_5f1f0296f570) {

auto const [slot_first5f1f0296f570, slot_second5f1f0296f570] = SLOT_5f1f0296f570;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5f1f0296f690, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5f1f0295b9f0(uint64_t* BUF_5f1f0296f570, uint64_t* BUF_5f1f0296f690, uint64_t* BUF_IDX_5f1f0296f690, HASHTABLE_PROBE HT_5f1f0296f570, HASHTABLE_INSERT HT_5f1f0296f690, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_5f1f0296f570 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_5f1f0296f570 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_5f1f0296f570.for_each(KEY_5f1f0296f570, [&] __device__ (auto const SLOT_5f1f0296f570) {
auto const [slot_first5f1f0296f570, slot_second5f1f0296f570] = SLOT_5f1f0296f570;
if (!(true)) return;
uint64_t KEY_5f1f0296f690 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5f1f0296f690 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5f1f0296f690 = atomicAdd((int*)BUF_IDX_5f1f0296f690, 1);
HT_5f1f0296f690.insert(cuco::pair{KEY_5f1f0296f690, buf_idx_5f1f0296f690});
BUF_5f1f0296f690[buf_idx_5f1f0296f690 * 2 + 0] = tid;
BUF_5f1f0296f690[buf_idx_5f1f0296f690 * 2 + 1] = BUF_5f1f0296f570[slot_second5f1f0296f570 * 1 + 0];
});
}
__global__ void count_5f1f02987af0(uint64_t* COUNT5f1f02977190, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
//Materialize count
atomicAdd((int*)COUNT5f1f02977190, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5f1f02987af0(uint64_t* BUF_5f1f02977190, uint64_t* BUF_IDX_5f1f02977190, HASHTABLE_INSERT HT_5f1f02977190, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_5f1f02977190 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_5f1f02977190 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_5f1f02977190 = atomicAdd((int*)BUF_IDX_5f1f02977190, 1);
HT_5f1f02977190.insert(cuco::pair{KEY_5f1f02977190, buf_idx_5f1f02977190});
BUF_5f1f02977190[buf_idx_5f1f02977190 * 1 + 0] = tid;
}
__global__ void count_5f1f0295b3c0(uint64_t* COUNT5f1f02977250, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
//Materialize count
atomicAdd((int*)COUNT5f1f02977250, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5f1f0295b3c0(uint64_t* BUF_5f1f02977250, uint64_t* BUF_IDX_5f1f02977250, HASHTABLE_INSERT HT_5f1f02977250, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
uint64_t KEY_5f1f02977250 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_5f1f02977250 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_5f1f02977250 = atomicAdd((int*)BUF_IDX_5f1f02977250, 1);
HT_5f1f02977250.insert(cuco::pair{KEY_5f1f02977250, buf_idx_5f1f02977250});
BUF_5f1f02977250[buf_idx_5f1f02977250 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5f1f0297f920(uint64_t* BUF_5f1f0296f690, uint64_t* COUNT5f1f02977310, HASHTABLE_PROBE HT_5f1f0296f690, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= partsupp_size) return;
uint64_t KEY_5f1f0296f690 = 0;
auto reg_partsupp__ps_suppkey = partsupp__ps_suppkey[tid];

KEY_5f1f0296f690 |= reg_partsupp__ps_suppkey;
//Probe Hash table
HT_5f1f0296f690.for_each(KEY_5f1f0296f690, [&] __device__ (auto const SLOT_5f1f0296f690) {

auto const [slot_first5f1f0296f690, slot_second5f1f0296f690] = SLOT_5f1f0296f690;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5f1f02977310, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5f1f0297f920(uint64_t* BUF_5f1f0296f690, uint64_t* BUF_5f1f02977310, uint64_t* BUF_IDX_5f1f02977310, HASHTABLE_PROBE HT_5f1f0296f690, HASHTABLE_INSERT HT_5f1f02977310, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size, DBI32Type* supplier__s_suppkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= partsupp_size) return;
uint64_t KEY_5f1f0296f690 = 0;
auto reg_partsupp__ps_suppkey = partsupp__ps_suppkey[tid];

KEY_5f1f0296f690 |= reg_partsupp__ps_suppkey;
//Probe Hash table
HT_5f1f0296f690.for_each(KEY_5f1f0296f690, [&] __device__ (auto const SLOT_5f1f0296f690) {
auto const [slot_first5f1f0296f690, slot_second5f1f0296f690] = SLOT_5f1f0296f690;
if (!(true)) return;
uint64_t KEY_5f1f02977310 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[BUF_5f1f0296f690[slot_second5f1f0296f690 * 2 + 0]];

KEY_5f1f02977310 |= reg_supplier__s_suppkey;
auto reg_partsupp__ps_partkey = partsupp__ps_partkey[tid];
KEY_5f1f02977310 <<= 32;
KEY_5f1f02977310 |= reg_partsupp__ps_partkey;
// Insert hash table kernel;
auto buf_idx_5f1f02977310 = atomicAdd((int*)BUF_IDX_5f1f02977310, 1);
HT_5f1f02977310.insert(cuco::pair{KEY_5f1f02977310, buf_idx_5f1f02977310});
BUF_5f1f02977310[buf_idx_5f1f02977310 * 3 + 0] = BUF_5f1f0296f690[slot_second5f1f0296f690 * 2 + 0];
BUF_5f1f02977310[buf_idx_5f1f02977310 * 3 + 1] = tid;
BUF_5f1f02977310[buf_idx_5f1f02977310 * 3 + 2] = BUF_5f1f0296f690[slot_second5f1f0296f690 * 2 + 1];
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5f1f0297ce00(uint64_t* BUF_5f1f02977190, uint64_t* BUF_5f1f02977250, uint64_t* BUF_5f1f02977310, HASHTABLE_INSERT HT_5f1f0292c180, HASHTABLE_PROBE HT_5f1f02977190, HASHTABLE_PROBE HT_5f1f02977250, HASHTABLE_PROBE HT_5f1f02977310, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded, DBDateType* orders__o_orderdate) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_5f1f02977190 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5f1f02977190 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5f1f02977190.for_each(KEY_5f1f02977190, [&] __device__ (auto const SLOT_5f1f02977190) {

auto const [slot_first5f1f02977190, slot_second5f1f02977190] = SLOT_5f1f02977190;
if (!(true)) return;
uint64_t KEY_5f1f02977250 = 0;
auto reg_lineitem__l_partkey = lineitem__l_partkey[tid];

KEY_5f1f02977250 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_5f1f02977250.for_each(KEY_5f1f02977250, [&] __device__ (auto const SLOT_5f1f02977250) {

auto const [slot_first5f1f02977250, slot_second5f1f02977250] = SLOT_5f1f02977250;
if (!(true)) return;
uint64_t KEY_5f1f02977310 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_5f1f02977310 |= reg_lineitem__l_suppkey;
KEY_5f1f02977310 <<= 32;
KEY_5f1f02977310 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_5f1f02977310.for_each(KEY_5f1f02977310, [&] __device__ (auto const SLOT_5f1f02977310) {

auto const [slot_first5f1f02977310, slot_second5f1f02977310] = SLOT_5f1f02977310;
if (!(true)) return;
uint64_t KEY_5f1f0292c180 = 0;
auto reg_nation__n_name_encoded = nation__n_name_encoded[BUF_5f1f02977310[slot_second5f1f02977310 * 3 + 2]];

KEY_5f1f0292c180 |= reg_nation__n_name_encoded;
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_5f1f02977190[slot_second5f1f02977190 * 1 + 0]];
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_orders__o_orderdate);
KEY_5f1f0292c180 <<= 32;
KEY_5f1f0292c180 |= (DBI32Type)reg_map0__tmp_attr0;
//Create aggregation hash table
HT_5f1f0292c180.insert(cuco::pair{KEY_5f1f0292c180, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5f1f0297ce00(uint64_t* BUF_5f1f02977190, uint64_t* BUF_5f1f02977250, uint64_t* BUF_5f1f02977310, HASHTABLE_FIND HT_5f1f0292c180, HASHTABLE_PROBE HT_5f1f02977190, HASHTABLE_PROBE HT_5f1f02977250, HASHTABLE_PROBE HT_5f1f02977310, DBI64Type* KEY_5f1f0292c180map0__tmp_attr0, DBI16Type* KEY_5f1f0292c180nation__n_name_encoded, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded, DBDateType* orders__o_orderdate, DBDecimalType* partsupp__ps_supplycost) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_5f1f02977190 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5f1f02977190 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5f1f02977190.for_each(KEY_5f1f02977190, [&] __device__ (auto const SLOT_5f1f02977190) {
auto const [slot_first5f1f02977190, slot_second5f1f02977190] = SLOT_5f1f02977190;
if (!(true)) return;
uint64_t KEY_5f1f02977250 = 0;
auto reg_lineitem__l_partkey = lineitem__l_partkey[tid];

KEY_5f1f02977250 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_5f1f02977250.for_each(KEY_5f1f02977250, [&] __device__ (auto const SLOT_5f1f02977250) {
auto const [slot_first5f1f02977250, slot_second5f1f02977250] = SLOT_5f1f02977250;
if (!(true)) return;
uint64_t KEY_5f1f02977310 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_5f1f02977310 |= reg_lineitem__l_suppkey;
KEY_5f1f02977310 <<= 32;
KEY_5f1f02977310 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_5f1f02977310.for_each(KEY_5f1f02977310, [&] __device__ (auto const SLOT_5f1f02977310) {
auto const [slot_first5f1f02977310, slot_second5f1f02977310] = SLOT_5f1f02977310;
if (!(true)) return;
uint64_t KEY_5f1f0292c180 = 0;
auto reg_nation__n_name_encoded = nation__n_name_encoded[BUF_5f1f02977310[slot_second5f1f02977310 * 3 + 2]];

KEY_5f1f0292c180 |= reg_nation__n_name_encoded;
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_5f1f02977190[slot_second5f1f02977190 * 1 + 0]];
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_orders__o_orderdate);
KEY_5f1f0292c180 <<= 32;
KEY_5f1f0292c180 |= (DBI32Type)reg_map0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_5f1f0292c180 = HT_5f1f0292c180.find(KEY_5f1f0292c180)->second;
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
auto reg_partsupp__ps_supplycost = partsupp__ps_supplycost[BUF_5f1f02977310[slot_second5f1f02977310 * 3 + 1]];
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = ((reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount))) - ((reg_partsupp__ps_supplycost) * (reg_lineitem__l_quantity));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_5f1f0292c180], reg_map0__tmp_attr1);
KEY_5f1f0292c180nation__n_name_encoded[buf_idx_5f1f0292c180] = reg_nation__n_name_encoded;
KEY_5f1f0292c180map0__tmp_attr0[buf_idx_5f1f0292c180] = reg_map0__tmp_attr0;
});
});
});
}
__global__ void count_5f1f02996ab0(size_t COUNT5f1f0292c180, uint64_t* COUNT5f1f0293eb10) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5f1f0292c180) return;
//Materialize count
atomicAdd((int*)COUNT5f1f0293eb10, 1);
}
__global__ void main_5f1f02996ab0(size_t COUNT5f1f0292c180, DBDecimalType* MAT5f1f0293eb10aggr0__tmp_attr2, DBI64Type* MAT5f1f0293eb10map0__tmp_attr0, DBI16Type* MAT5f1f0293eb10nation__n_name_encoded, uint64_t* MAT_IDX5f1f0293eb10, DBDecimalType* aggr0__tmp_attr2, DBI64Type* map0__tmp_attr0, DBI16Type* nation__n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5f1f0292c180) return;
//Materialize buffers
auto mat_idx5f1f0293eb10 = atomicAdd((int*)MAT_IDX5f1f0293eb10, 1);
auto reg_nation__n_name_encoded = nation__n_name_encoded[tid];
MAT5f1f0293eb10nation__n_name_encoded[mat_idx5f1f0293eb10] = reg_nation__n_name_encoded;
auto reg_map0__tmp_attr0 = map0__tmp_attr0[tid];
MAT5f1f0293eb10map0__tmp_attr0[mat_idx5f1f0293eb10] = reg_map0__tmp_attr0;
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT5f1f0293eb10aggr0__tmp_attr2[mat_idx5f1f0293eb10] = reg_aggr0__tmp_attr2;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map) {
//Materialize count
uint64_t* d_COUNT5f1f0296f570;
hipMalloc(&d_COUNT5f1f0296f570, sizeof(uint64_t));
hipMemset(d_COUNT5f1f0296f570, 0, sizeof(uint64_t));
count_5f1f02980ad0<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT5f1f0296f570, nation_size);
uint64_t COUNT5f1f0296f570;
hipMemcpy(&COUNT5f1f0296f570, d_COUNT5f1f0296f570, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5f1f0296f570;
hipMalloc(&d_BUF_IDX_5f1f0296f570, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5f1f0296f570, 0, sizeof(uint64_t));
uint64_t* d_BUF_5f1f0296f570;
hipMalloc(&d_BUF_5f1f0296f570, sizeof(uint64_t) * COUNT5f1f0296f570 * 1);
auto d_HT_5f1f0296f570 = cuco::experimental::static_multimap{ (int)COUNT5f1f0296f570*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5f1f02980ad0<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_5f1f0296f570, d_BUF_IDX_5f1f0296f570, d_HT_5f1f0296f570.ref(cuco::insert), d_nation__n_nationkey, nation_size);
//Materialize count
uint64_t* d_COUNT5f1f0296f690;
hipMalloc(&d_COUNT5f1f0296f690, sizeof(uint64_t));
hipMemset(d_COUNT5f1f0296f690, 0, sizeof(uint64_t));
count_5f1f0295b9f0<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_5f1f0296f570, d_COUNT5f1f0296f690, d_HT_5f1f0296f570.ref(cuco::for_each), d_supplier__s_nationkey, supplier_size);
uint64_t COUNT5f1f0296f690;
hipMemcpy(&COUNT5f1f0296f690, d_COUNT5f1f0296f690, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5f1f0296f690;
hipMalloc(&d_BUF_IDX_5f1f0296f690, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5f1f0296f690, 0, sizeof(uint64_t));
uint64_t* d_BUF_5f1f0296f690;
hipMalloc(&d_BUF_5f1f0296f690, sizeof(uint64_t) * COUNT5f1f0296f690 * 2);
auto d_HT_5f1f0296f690 = cuco::experimental::static_multimap{ (int)COUNT5f1f0296f690*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5f1f0295b9f0<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_5f1f0296f570, d_BUF_5f1f0296f690, d_BUF_IDX_5f1f0296f690, d_HT_5f1f0296f570.ref(cuco::for_each), d_HT_5f1f0296f690.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5f1f02977190;
hipMalloc(&d_COUNT5f1f02977190, sizeof(uint64_t));
hipMemset(d_COUNT5f1f02977190, 0, sizeof(uint64_t));
count_5f1f02987af0<<<std::ceil((float)orders_size/32.), 32>>>(d_COUNT5f1f02977190, orders_size);
uint64_t COUNT5f1f02977190;
hipMemcpy(&COUNT5f1f02977190, d_COUNT5f1f02977190, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5f1f02977190;
hipMalloc(&d_BUF_IDX_5f1f02977190, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5f1f02977190, 0, sizeof(uint64_t));
uint64_t* d_BUF_5f1f02977190;
hipMalloc(&d_BUF_5f1f02977190, sizeof(uint64_t) * COUNT5f1f02977190 * 1);
auto d_HT_5f1f02977190 = cuco::experimental::static_multimap{ (int)COUNT5f1f02977190*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5f1f02987af0<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5f1f02977190, d_BUF_IDX_5f1f02977190, d_HT_5f1f02977190.ref(cuco::insert), d_orders__o_orderkey, orders_size);
//Materialize count
uint64_t* d_COUNT5f1f02977250;
hipMalloc(&d_COUNT5f1f02977250, sizeof(uint64_t));
hipMemset(d_COUNT5f1f02977250, 0, sizeof(uint64_t));
count_5f1f0295b3c0<<<std::ceil((float)part_size/32.), 32>>>(d_COUNT5f1f02977250, part_size);
uint64_t COUNT5f1f02977250;
hipMemcpy(&COUNT5f1f02977250, d_COUNT5f1f02977250, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5f1f02977250;
hipMalloc(&d_BUF_IDX_5f1f02977250, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5f1f02977250, 0, sizeof(uint64_t));
uint64_t* d_BUF_5f1f02977250;
hipMalloc(&d_BUF_5f1f02977250, sizeof(uint64_t) * COUNT5f1f02977250 * 1);
auto d_HT_5f1f02977250 = cuco::experimental::static_multimap{ (int)COUNT5f1f02977250*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5f1f0295b3c0<<<std::ceil((float)part_size/32.), 32>>>(d_BUF_5f1f02977250, d_BUF_IDX_5f1f02977250, d_HT_5f1f02977250.ref(cuco::insert), d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT5f1f02977310;
hipMalloc(&d_COUNT5f1f02977310, sizeof(uint64_t));
hipMemset(d_COUNT5f1f02977310, 0, sizeof(uint64_t));
count_5f1f0297f920<<<std::ceil((float)partsupp_size/32.), 32>>>(d_BUF_5f1f0296f690, d_COUNT5f1f02977310, d_HT_5f1f0296f690.ref(cuco::for_each), d_partsupp__ps_suppkey, partsupp_size);
uint64_t COUNT5f1f02977310;
hipMemcpy(&COUNT5f1f02977310, d_COUNT5f1f02977310, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5f1f02977310;
hipMalloc(&d_BUF_IDX_5f1f02977310, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5f1f02977310, 0, sizeof(uint64_t));
uint64_t* d_BUF_5f1f02977310;
hipMalloc(&d_BUF_5f1f02977310, sizeof(uint64_t) * COUNT5f1f02977310 * 3);
auto d_HT_5f1f02977310 = cuco::experimental::static_multimap{ (int)COUNT5f1f02977310*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5f1f0297f920<<<std::ceil((float)partsupp_size/32.), 32>>>(d_BUF_5f1f0296f690, d_BUF_5f1f02977310, d_BUF_IDX_5f1f02977310, d_HT_5f1f0296f690.ref(cuco::for_each), d_HT_5f1f02977310.ref(cuco::insert), d_partsupp__ps_partkey, d_partsupp__ps_suppkey, partsupp_size, d_supplier__s_suppkey);
//Create aggregation hash table
auto d_HT_5f1f0292c180 = cuco::static_map{ (int)48009721*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5f1f0297ce00<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5f1f02977190, d_BUF_5f1f02977250, d_BUF_5f1f02977310, d_HT_5f1f0292c180.ref(cuco::insert), d_HT_5f1f02977190.ref(cuco::for_each), d_HT_5f1f02977250.ref(cuco::for_each), d_HT_5f1f02977310.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded, d_orders__o_orderdate);
size_t COUNT5f1f0292c180 = d_HT_5f1f0292c180.size();
thrust::device_vector<int64_t> keys_5f1f0292c180(COUNT5f1f0292c180), vals_5f1f0292c180(COUNT5f1f0292c180);
d_HT_5f1f0292c180.retrieve_all(keys_5f1f0292c180.begin(), vals_5f1f0292c180.begin());
d_HT_5f1f0292c180.clear();
int64_t* raw_keys5f1f0292c180 = thrust::raw_pointer_cast(keys_5f1f0292c180.data());
insertKeys<<<std::ceil((float)COUNT5f1f0292c180/32.), 32>>>(raw_keys5f1f0292c180, d_HT_5f1f0292c180.ref(cuco::insert), COUNT5f1f0292c180);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT5f1f0292c180);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT5f1f0292c180);
DBI16Type* d_KEY_5f1f0292c180nation__n_name_encoded;
hipMalloc(&d_KEY_5f1f0292c180nation__n_name_encoded, sizeof(DBI16Type) * COUNT5f1f0292c180);
hipMemset(d_KEY_5f1f0292c180nation__n_name_encoded, 0, sizeof(DBI16Type) * COUNT5f1f0292c180);
DBI64Type* d_KEY_5f1f0292c180map0__tmp_attr0;
hipMalloc(&d_KEY_5f1f0292c180map0__tmp_attr0, sizeof(DBI64Type) * COUNT5f1f0292c180);
hipMemset(d_KEY_5f1f0292c180map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT5f1f0292c180);
main_5f1f0297ce00<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5f1f02977190, d_BUF_5f1f02977250, d_BUF_5f1f02977310, d_HT_5f1f0292c180.ref(cuco::find), d_HT_5f1f02977190.ref(cuco::for_each), d_HT_5f1f02977250.ref(cuco::for_each), d_HT_5f1f02977310.ref(cuco::for_each), d_KEY_5f1f0292c180map0__tmp_attr0, d_KEY_5f1f0292c180nation__n_name_encoded, d_aggr0__tmp_attr2, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_quantity, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded, d_orders__o_orderdate, d_partsupp__ps_supplycost);
//Materialize count
uint64_t* d_COUNT5f1f0293eb10;
hipMalloc(&d_COUNT5f1f0293eb10, sizeof(uint64_t));
hipMemset(d_COUNT5f1f0293eb10, 0, sizeof(uint64_t));
count_5f1f02996ab0<<<std::ceil((float)COUNT5f1f0292c180/32.), 32>>>(COUNT5f1f0292c180, d_COUNT5f1f0293eb10);
uint64_t COUNT5f1f0293eb10;
hipMemcpy(&COUNT5f1f0293eb10, d_COUNT5f1f0293eb10, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5f1f0293eb10;
hipMalloc(&d_MAT_IDX5f1f0293eb10, sizeof(uint64_t));
hipMemset(d_MAT_IDX5f1f0293eb10, 0, sizeof(uint64_t));
auto MAT5f1f0293eb10nation__n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5f1f0293eb10);
DBI16Type* d_MAT5f1f0293eb10nation__n_name_encoded;
hipMalloc(&d_MAT5f1f0293eb10nation__n_name_encoded, sizeof(DBI16Type) * COUNT5f1f0293eb10);
auto MAT5f1f0293eb10map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT5f1f0293eb10);
DBI64Type* d_MAT5f1f0293eb10map0__tmp_attr0;
hipMalloc(&d_MAT5f1f0293eb10map0__tmp_attr0, sizeof(DBI64Type) * COUNT5f1f0293eb10);
auto MAT5f1f0293eb10aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5f1f0293eb10);
DBDecimalType* d_MAT5f1f0293eb10aggr0__tmp_attr2;
hipMalloc(&d_MAT5f1f0293eb10aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT5f1f0293eb10);
main_5f1f02996ab0<<<std::ceil((float)COUNT5f1f0292c180/32.), 32>>>(COUNT5f1f0292c180, d_MAT5f1f0293eb10aggr0__tmp_attr2, d_MAT5f1f0293eb10map0__tmp_attr0, d_MAT5f1f0293eb10nation__n_name_encoded, d_MAT_IDX5f1f0293eb10, d_aggr0__tmp_attr2, d_KEY_5f1f0292c180map0__tmp_attr0, d_KEY_5f1f0292c180nation__n_name_encoded);
hipMemcpy(MAT5f1f0293eb10nation__n_name_encoded, d_MAT5f1f0293eb10nation__n_name_encoded, sizeof(DBI16Type) * COUNT5f1f0293eb10, hipMemcpyDeviceToHost);
hipMemcpy(MAT5f1f0293eb10map0__tmp_attr0, d_MAT5f1f0293eb10map0__tmp_attr0, sizeof(DBI64Type) * COUNT5f1f0293eb10, hipMemcpyDeviceToHost);
hipMemcpy(MAT5f1f0293eb10aggr0__tmp_attr2, d_MAT5f1f0293eb10aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT5f1f0293eb10, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5f1f0293eb10; i++) { std::cout << nation__n_name_map[MAT5f1f0293eb10nation__n_name_encoded[i]] << "\t";
std::cout << MAT5f1f0293eb10map0__tmp_attr0[i] << "\t";
std::cout << MAT5f1f0293eb10aggr0__tmp_attr2[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_5f1f0296f570);
hipFree(d_BUF_IDX_5f1f0296f570);
hipFree(d_COUNT5f1f0296f570);
hipFree(d_BUF_5f1f0296f690);
hipFree(d_BUF_IDX_5f1f0296f690);
hipFree(d_COUNT5f1f0296f690);
hipFree(d_BUF_5f1f02977190);
hipFree(d_BUF_IDX_5f1f02977190);
hipFree(d_COUNT5f1f02977190);
hipFree(d_BUF_5f1f02977250);
hipFree(d_BUF_IDX_5f1f02977250);
hipFree(d_COUNT5f1f02977250);
hipFree(d_BUF_5f1f02977310);
hipFree(d_BUF_IDX_5f1f02977310);
hipFree(d_COUNT5f1f02977310);
hipFree(d_KEY_5f1f0292c180map0__tmp_attr0);
hipFree(d_KEY_5f1f0292c180nation__n_name_encoded);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_COUNT5f1f0293eb10);
hipFree(d_MAT5f1f0293eb10aggr0__tmp_attr2);
hipFree(d_MAT5f1f0293eb10map0__tmp_attr0);
hipFree(d_MAT5f1f0293eb10nation__n_name_encoded);
hipFree(d_MAT_IDX5f1f0293eb10);
free(MAT5f1f0293eb10aggr0__tmp_attr2);
free(MAT5f1f0293eb10map0__tmp_attr0);
free(MAT5f1f0293eb10nation__n_name_encoded);
}