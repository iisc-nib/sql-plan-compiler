#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_INSERT HT_2, int64_t* cycles_per_warp_main_3_join_build_2, DBI32Type* part__p_partkey, size_t part_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_2.insert(cuco::pair{KEY_2[ITEM], ITEM*TB + tid});
BUF_2[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_build_2[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_1(uint64_t* BUF_2, HASHTABLE_PROBE HT_2, HASHTABLE_FIND HT_6, int* SLOT_COUNT_6, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr0__tmp_attr2, int64_t* cycles_per_warp_main_1_aggregation_6, int64_t* cycles_per_warp_main_1_join_probe_2, int64_t* cycles_per_warp_main_1_map_4, int64_t* cycles_per_warp_main_1_map_5, int64_t* cycles_per_warp_main_1_selection_0, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_partkey, DBDateType* lineitem__l_shipdate, size_t lineitem_size, DBStringType* part__p_type) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBDateType reg_lineitem__l_shipdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipdate[ITEM] = lineitem__l_shipdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 9374, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 9404, Predicate::lt);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
//Probe Hash table
int64_t slot_second2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_2 = HT_2.find(KEY_2[ITEM]);
if (SLOT_2 == HT_2.end()) {selection_flags[ITEM] = 0; continue;}
slot_second2[ITEM] = SLOT_2->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_probe_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_map_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
DBStringType reg_part__p_type[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_type[ITEM] = part__p_type[BUF_2[slot_second2[ITEM] * 1 + 0]];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_map_5[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr3[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr3[ITEM] = (reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]));
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = ((Like(reg_part__p_type[ITEM], "PROMO", "", nullptr, nullptr, 0))) ? ((reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]))) : (0.0);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_6 = get_aggregation_slot(KEY_6[ITEM], HT_6, SLOT_COUNT_6);
aggregate_sum(&aggr0__tmp_attr2[buf_idx_6], reg_map0__tmp_attr3[ITEM]);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_6], reg_map0__tmp_attr1[ITEM]);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_aggregation_6[blockIdx.x] = cycles_per_warp;}
}
__global__ void main_8(size_t COUNT6, DBDecimalType* MAT9map1__tmp_attr4, uint64_t* MAT_IDX9, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr0__tmp_attr2, int64_t* cycles_per_warp_main_8_map_7, int64_t* cycles_per_warp_main_8_materialize_9) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_8_map_7[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBDecimalType reg_aggr0__tmp_attr2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT6); ++ITEM) {
reg_aggr0__tmp_attr2[ITEM] = aggr0__tmp_attr2[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT6); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
DBDecimalType reg_map1__tmp_attr4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT6); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map1__tmp_attr4[ITEM] = ((100.00) * (reg_aggr0__tmp_attr0[ITEM])) / (reg_aggr0__tmp_attr2[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT6); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx9 = atomicAdd((int*)MAT_IDX9, 1);
MAT9map1__tmp_attr4[mat_idx9] = reg_map1__tmp_attr4[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_8_materialize_9[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_3_join_build_2;
auto main_3_join_build_2_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_build_2, sizeof(int64_t) * main_3_join_build_2_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_build_2, -1, sizeof(int64_t) * main_3_join_build_2_cpw_size);
size_t COUNT2 = part_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 1);
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)part_size/(float)TILE_SIZE), TB>>>(d_BUF_2, d_BUF_IDX_2, d_HT_2.ref(cuco::insert), d_cycles_per_warp_main_3_join_build_2, d_part__p_partkey, part_size);
int64_t* cycles_per_warp_main_3_join_build_2 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_build_2_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_build_2, d_cycles_per_warp_main_3_join_build_2, sizeof(int64_t) * main_3_join_build_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_build_2 ";
for (auto i=0ull; i < main_3_join_build_2_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_build_2[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_join_probe_2;
auto main_1_join_probe_2_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_probe_2, sizeof(int64_t) * main_1_join_probe_2_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_probe_2, -1, sizeof(int64_t) * main_1_join_probe_2_cpw_size);
int64_t* d_cycles_per_warp_main_1_map_4;
auto main_1_map_4_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_map_4, sizeof(int64_t) * main_1_map_4_cpw_size);
hipMemset(d_cycles_per_warp_main_1_map_4, -1, sizeof(int64_t) * main_1_map_4_cpw_size);
int64_t* d_cycles_per_warp_main_1_map_5;
auto main_1_map_5_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_map_5, sizeof(int64_t) * main_1_map_5_cpw_size);
hipMemset(d_cycles_per_warp_main_1_map_5, -1, sizeof(int64_t) * main_1_map_5_cpw_size);
int64_t* d_cycles_per_warp_main_1_aggregation_6;
auto main_1_aggregation_6_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_aggregation_6, sizeof(int64_t) * main_1_aggregation_6_cpw_size);
hipMemset(d_cycles_per_warp_main_1_aggregation_6, -1, sizeof(int64_t) * main_1_aggregation_6_cpw_size);
size_t COUNT6 = 1;
auto d_HT_6 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_6;
hipMalloc(&d_SLOT_COUNT_6, sizeof(int));
hipMemset(d_SLOT_COUNT_6, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT6);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT6);
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT6);
main_1<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TB>>>(d_BUF_2, d_HT_2.ref(cuco::find), d_HT_6.ref(cuco::insert_and_find), d_SLOT_COUNT_6, d_aggr0__tmp_attr0, d_aggr0__tmp_attr2, d_cycles_per_warp_main_1_aggregation_6, d_cycles_per_warp_main_1_join_probe_2, d_cycles_per_warp_main_1_map_4, d_cycles_per_warp_main_1_map_5, d_cycles_per_warp_main_1_selection_0, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_partkey, d_lineitem__l_shipdate, lineitem_size, d_part__p_type);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_probe_2 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_probe_2_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_probe_2, d_cycles_per_warp_main_1_join_probe_2, sizeof(int64_t) * main_1_join_probe_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_probe_2 ";
for (auto i=0ull; i < main_1_join_probe_2_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_probe_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_map_4 = (int64_t*)malloc(sizeof(int64_t) * main_1_map_4_cpw_size);
hipMemcpy(cycles_per_warp_main_1_map_4, d_cycles_per_warp_main_1_map_4, sizeof(int64_t) * main_1_map_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_map_4 ";
for (auto i=0ull; i < main_1_map_4_cpw_size; i++) std::cout << cycles_per_warp_main_1_map_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_map_5 = (int64_t*)malloc(sizeof(int64_t) * main_1_map_5_cpw_size);
hipMemcpy(cycles_per_warp_main_1_map_5, d_cycles_per_warp_main_1_map_5, sizeof(int64_t) * main_1_map_5_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_map_5 ";
for (auto i=0ull; i < main_1_map_5_cpw_size; i++) std::cout << cycles_per_warp_main_1_map_5[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_aggregation_6 = (int64_t*)malloc(sizeof(int64_t) * main_1_aggregation_6_cpw_size);
hipMemcpy(cycles_per_warp_main_1_aggregation_6, d_cycles_per_warp_main_1_aggregation_6, sizeof(int64_t) * main_1_aggregation_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_aggregation_6 ";
for (auto i=0ull; i < main_1_aggregation_6_cpw_size; i++) std::cout << cycles_per_warp_main_1_aggregation_6[i] << " ";
std::cout << std::endl;
COUNT6 = d_HT_6.size();
int64_t* d_cycles_per_warp_main_8_map_7;
auto main_8_map_7_cpw_size = std::ceil((float)COUNT6/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_8_map_7, sizeof(int64_t) * main_8_map_7_cpw_size);
hipMemset(d_cycles_per_warp_main_8_map_7, -1, sizeof(int64_t) * main_8_map_7_cpw_size);
int64_t* d_cycles_per_warp_main_8_materialize_9;
auto main_8_materialize_9_cpw_size = std::ceil((float)COUNT6/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_8_materialize_9, sizeof(int64_t) * main_8_materialize_9_cpw_size);
hipMemset(d_cycles_per_warp_main_8_materialize_9, -1, sizeof(int64_t) * main_8_materialize_9_cpw_size);
size_t COUNT9 = COUNT6;
//Materialize buffers
uint64_t* d_MAT_IDX9;
hipMalloc(&d_MAT_IDX9, sizeof(uint64_t));
hipMemset(d_MAT_IDX9, 0, sizeof(uint64_t));
auto MAT9map1__tmp_attr4 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT9);
DBDecimalType* d_MAT9map1__tmp_attr4;
hipMalloc(&d_MAT9map1__tmp_attr4, sizeof(DBDecimalType) * COUNT9);
main_8<<<std::ceil((float)COUNT6/(float)TILE_SIZE), TB>>>(COUNT6, d_MAT9map1__tmp_attr4, d_MAT_IDX9, d_aggr0__tmp_attr0, d_aggr0__tmp_attr2, d_cycles_per_warp_main_8_map_7, d_cycles_per_warp_main_8_materialize_9);
uint64_t MATCOUNT_9 = 0;
hipMemcpy(&MATCOUNT_9, d_MAT_IDX9, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT9map1__tmp_attr4, d_MAT9map1__tmp_attr4, sizeof(DBDecimalType) * COUNT9, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_8_map_7 = (int64_t*)malloc(sizeof(int64_t) * main_8_map_7_cpw_size);
hipMemcpy(cycles_per_warp_main_8_map_7, d_cycles_per_warp_main_8_map_7, sizeof(int64_t) * main_8_map_7_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_8_map_7 ";
for (auto i=0ull; i < main_8_map_7_cpw_size; i++) std::cout << cycles_per_warp_main_8_map_7[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_8_materialize_9 = (int64_t*)malloc(sizeof(int64_t) * main_8_materialize_9_cpw_size);
hipMemcpy(cycles_per_warp_main_8_materialize_9, d_cycles_per_warp_main_8_materialize_9, sizeof(int64_t) * main_8_materialize_9_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_8_materialize_9 ";
for (auto i=0ull; i < main_8_materialize_9_cpw_size; i++) std::cout << cycles_per_warp_main_8_materialize_9[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_MAT9map1__tmp_attr4);
hipFree(d_MAT_IDX9);
free(MAT9map1__tmp_attr4);
}