#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_1(uint64_t* COUNT3, DBStringType* part__p_brand, DBStringType* part__p_container, size_t part_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_part__p_container[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_container[ITEM] = part__p_container[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq);
}
DBStringType reg_part__p_brand[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_brand[ITEM] = part__p_brand[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT3, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_3, uint64_t* BUF_IDX_3, HASHTABLE_INSERT HT_3, int64_t* cycles_per_warp_main_1_join_build_3, int64_t* cycles_per_warp_main_1_selection_0, int64_t* cycles_per_warp_main_1_selection_2, DBStringType* part__p_brand, DBStringType* part__p_container, DBI32Type* part__p_partkey, size_t part_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_part__p_container[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_container[ITEM] = part__p_container[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
DBStringType reg_part__p_brand[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_brand[ITEM] = part__p_brand[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_3[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_3[ITEM] = 0;
KEY_3[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_3 = atomicAdd((int*)BUF_IDX_3, 1);
HT_3.insert(cuco::pair{KEY_3[ITEM], buf_idx_3});
BUF_3[(buf_idx_3) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_build_3[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_4(uint64_t* BUF_3, HASHTABLE_PROBE HT_3, HASHTABLE_INSERT HT_5, size_t lineitem_size, DBI32Type* lineitem_u_1__l_partkey) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_3[ITEMS_PER_THREAD];
DBI32Type reg_lineitem_u_1__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem_u_1__l_partkey[ITEM] = lineitem_u_1__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_3[ITEM] = 0;
KEY_3[ITEM] |= reg_lineitem_u_1__l_partkey[ITEM];
}
int64_t slot_second3[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_3 = HT_3.find(KEY_3[ITEM]);
if (SLOT_3 == HT_3.end()) {selection_flags[ITEM] = 0; continue;}
slot_second3[ITEM] = SLOT_3->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_5[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_5[ITEM] = 0;
KEY_5[ITEM] |= reg_lineitem_u_1__l_partkey[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_5.insert(cuco::pair{KEY_5[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_4(uint64_t* BUF_3, HASHTABLE_PROBE HT_3, HASHTABLE_FIND HT_5, DBI32Type* KEY_5lineitem_u_1__l_partkey, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, int64_t* cycles_per_warp_main_4_aggregation_5, int64_t* cycles_per_warp_main_4_join_probe_3, size_t lineitem_size, DBI32Type* lineitem_u_1__l_partkey, DBDecimalType* lineitem_u_1__l_quantity, DBI32Type* moved_aggr__p_partkey, DBI32Type* part__p_partkey) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_3[ITEMS_PER_THREAD];
DBI32Type reg_lineitem_u_1__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem_u_1__l_partkey[ITEM] = lineitem_u_1__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_3[ITEM] = 0;
KEY_3[ITEM] |= reg_lineitem_u_1__l_partkey[ITEM];
}
int64_t slot_second3[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_3 = HT_3.find(KEY_3[ITEM]);
if (SLOT_3 == HT_3.end()) {selection_flags[ITEM] = 0; continue;}
slot_second3[ITEM] = SLOT_3->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_4_join_probe_3[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_5[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_5[ITEM] = 0;
KEY_5[ITEM] |= reg_lineitem_u_1__l_partkey[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem_u_1__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem_u_1__l_quantity[ITEM] = lineitem_u_1__l_quantity[ITEM*TB + tid];
}
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_partkey[ITEM] = part__p_partkey[BUF_3[slot_second3[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_5 = HT_5.find(KEY_5[ITEM])->second;
aggregate_sum(&aggr_rw__rw0[buf_idx_5], reg_lineitem_u_1__l_quantity[ITEM]);
aggregate_sum(&aggr_rw__rw1[buf_idx_5], 1);
aggregate_any(&moved_aggr__p_partkey[buf_idx_5], reg_part__p_partkey[ITEM]);
KEY_5lineitem_u_1__l_partkey[buf_idx_5] = reg_lineitem_u_1__l_partkey[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_4_aggregation_5[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_7(uint64_t* COUNT10, size_t COUNT5, DBI32Type* moved_aggr__p_partkey) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT5); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT5); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT10, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_7(uint64_t* BUF_10, uint64_t* BUF_IDX_10, size_t COUNT5, HASHTABLE_INSERT HT_10, int64_t* cycles_per_warp_main_7_join_build_10, int64_t* cycles_per_warp_main_7_map_6, int64_t* cycles_per_warp_main_7_map_8, int64_t* cycles_per_warp_main_7_selection_9, DBI32Type* moved_aggr__p_partkey) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_map_6[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_map_8[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT5); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_selection_9[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT5); ++ITEM) {
reg_part__p_partkey[ITEM] = moved_aggr__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT5); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT5); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_10 = atomicAdd((int*)BUF_IDX_10, 1);
HT_10.insert(cuco::pair{KEY_10[ITEM], buf_idx_10});
BUF_10[(buf_idx_10) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_build_10[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_11(uint64_t* BUF_10, HASHTABLE_PROBE HT_10, HASHTABLE_INSERT HT_12, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, size_t lineitem_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
int64_t slot_second10[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_10 = HT_10.find(KEY_10[ITEM]);
if (SLOT_10 == HT_10.end()) {selection_flags[ITEM] = 0; continue;}
slot_second10[ITEM] = SLOT_10->second;
}
DBDecimalType reg_lineitem__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_quantity[ITEM] = lineitem__l_quantity[ITEM*TB + tid];
}
DBI64Type reg_aggr_rw__rw1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr_rw__rw1[ITEM] = aggr_rw__rw1[BUF_10[slot_second10[ITEM] * 1 + 0]];
}
DBDecimalType reg_aggr_rw__rw0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr_rw__rw0[ITEM] = aggr_rw__rw0[BUF_10[slot_second10[ITEM] * 1 + 0]];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr0__tmp_attr0[ITEM] = (reg_aggr_rw__rw0[ITEM]) / ((DBDecimalType)(reg_aggr_rw__rw1[ITEM]));
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (0.2) * (reg_aggr0__tmp_attr0[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(((DBDecimalType)reg_lineitem__l_quantity[ITEM]), reg_map0__tmp_attr1[ITEM], Predicate::lt)) && (true);
}
uint64_t KEY_12[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_12[ITEM] = 0;
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_12.insert(cuco::pair{KEY_12[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_11(uint64_t* BUF_10, HASHTABLE_PROBE HT_10, HASHTABLE_FIND HT_12, DBDecimalType* aggr1__tmp_attr2, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, int64_t* cycles_per_warp_main_11_aggregation_12, int64_t* cycles_per_warp_main_11_join_probe_10, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, size_t lineitem_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
int64_t slot_second10[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_10 = HT_10.find(KEY_10[ITEM]);
if (SLOT_10 == HT_10.end()) {selection_flags[ITEM] = 0; continue;}
slot_second10[ITEM] = SLOT_10->second;
}
DBDecimalType reg_lineitem__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_quantity[ITEM] = lineitem__l_quantity[ITEM*TB + tid];
}
DBI64Type reg_aggr_rw__rw1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr_rw__rw1[ITEM] = aggr_rw__rw1[BUF_10[slot_second10[ITEM] * 1 + 0]];
}
DBDecimalType reg_aggr_rw__rw0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr_rw__rw0[ITEM] = aggr_rw__rw0[BUF_10[slot_second10[ITEM] * 1 + 0]];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr0__tmp_attr0[ITEM] = (reg_aggr_rw__rw0[ITEM]) / ((DBDecimalType)(reg_aggr_rw__rw1[ITEM]));
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (0.2) * (reg_aggr0__tmp_attr0[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(((DBDecimalType)reg_lineitem__l_quantity[ITEM]), reg_map0__tmp_attr1[ITEM], Predicate::lt)) && (true);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_join_probe_10[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_12[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_12[ITEM] = 0;
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_12 = HT_12.find(KEY_12[ITEM])->second;
aggregate_sum(&aggr1__tmp_attr2[buf_idx_12], reg_lineitem__l_extendedprice[ITEM]);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_aggregation_12[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_14(size_t COUNT12, uint64_t* COUNT15) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT12); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT15, 1);
}
}
__global__ void main_14(size_t COUNT12, DBDecimalType* MAT15map1__tmp_attr3, uint64_t* MAT_IDX15, DBDecimalType* aggr1__tmp_attr2, int64_t* cycles_per_warp_main_14_map_13, int64_t* cycles_per_warp_main_14_materialize_15) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_14_map_13[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBDecimalType reg_aggr1__tmp_attr2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT12); ++ITEM) {
reg_aggr1__tmp_attr2[ITEM] = aggr1__tmp_attr2[ITEM*TB + tid];
}
DBDecimalType reg_map1__tmp_attr3[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT12); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map1__tmp_attr3[ITEM] = (reg_aggr1__tmp_attr2[ITEM]) / (7.0);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT12); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx15 = atomicAdd((int*)MAT_IDX15, 1);
MAT15map1__tmp_attr3[mat_idx15] = reg_map1__tmp_attr3[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_14_materialize_15[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_2;
auto main_1_selection_2_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_2, -1, sizeof(int64_t) * main_1_selection_2_cpw_size);
//Materialize count
uint64_t* d_COUNT3;
hipMalloc(&d_COUNT3, sizeof(uint64_t));
hipMemset(d_COUNT3, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)part_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT3, d_part__p_brand, d_part__p_container, part_size);
uint64_t COUNT3;
hipMemcpy(&COUNT3, d_COUNT3, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_1_join_build_3;
auto main_1_join_build_3_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_build_3, sizeof(int64_t) * main_1_join_build_3_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_build_3, -1, sizeof(int64_t) * main_1_join_build_3_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_3;
hipMalloc(&d_BUF_IDX_3, sizeof(uint64_t));
hipMemset(d_BUF_IDX_3, 0, sizeof(uint64_t));
uint64_t* d_BUF_3;
hipMalloc(&d_BUF_3, sizeof(uint64_t) * COUNT3 * 1);
auto d_HT_3 = cuco::static_map{ (int)COUNT3*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)part_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_3, d_BUF_IDX_3, d_HT_3.ref(cuco::insert), d_cycles_per_warp_main_1_join_build_3, d_cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_2, d_part__p_brand, d_part__p_container, d_part__p_partkey, part_size);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_2 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_2 ";
for (auto i=0ull; i < main_1_selection_2_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_build_3 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_build_3_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_build_3, d_cycles_per_warp_main_1_join_build_3, sizeof(int64_t) * main_1_join_build_3_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_build_3 ";
for (auto i=0ull; i < main_1_join_build_3_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_build_3[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_4_join_probe_3;
auto main_4_join_probe_3_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_4_join_probe_3, sizeof(int64_t) * main_4_join_probe_3_cpw_size);
hipMemset(d_cycles_per_warp_main_4_join_probe_3, -1, sizeof(int64_t) * main_4_join_probe_3_cpw_size);
//Create aggregation hash table
auto d_HT_5 = cuco::static_map{ (int)6001215*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_4<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_3, d_HT_3.ref(cuco::find), d_HT_5.ref(cuco::insert), lineitem_size, d_lineitem__l_partkey);
size_t COUNT5 = d_HT_5.size();
thrust::device_vector<int64_t> keys_5(COUNT5), vals_5(COUNT5);
d_HT_5.retrieve_all(keys_5.begin(), vals_5.begin());
d_HT_5.clear();
int64_t* raw_keys5 = thrust::raw_pointer_cast(keys_5.data());
insertKeys<<<std::ceil((float)COUNT5/128.), 128>>>(raw_keys5, d_HT_5.ref(cuco::insert), COUNT5);
int64_t* d_cycles_per_warp_main_4_aggregation_5;
auto main_4_aggregation_5_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_4_aggregation_5, sizeof(int64_t) * main_4_aggregation_5_cpw_size);
hipMemset(d_cycles_per_warp_main_4_aggregation_5, -1, sizeof(int64_t) * main_4_aggregation_5_cpw_size);
//Aggregate in hashtable
DBDecimalType* d_aggr_rw__rw0;
hipMalloc(&d_aggr_rw__rw0, sizeof(DBDecimalType) * COUNT5);
hipMemset(d_aggr_rw__rw0, 0, sizeof(DBDecimalType) * COUNT5);
DBI64Type* d_aggr_rw__rw1;
hipMalloc(&d_aggr_rw__rw1, sizeof(DBI64Type) * COUNT5);
hipMemset(d_aggr_rw__rw1, 0, sizeof(DBI64Type) * COUNT5);
DBI32Type* d_moved_aggr__p_partkey;
hipMalloc(&d_moved_aggr__p_partkey, sizeof(DBI32Type) * COUNT5);
hipMemset(d_moved_aggr__p_partkey, 0, sizeof(DBI32Type) * COUNT5);
DBI32Type* d_KEY_5lineitem_u_1__l_partkey;
hipMalloc(&d_KEY_5lineitem_u_1__l_partkey, sizeof(DBI32Type) * COUNT5);
hipMemset(d_KEY_5lineitem_u_1__l_partkey, 0, sizeof(DBI32Type) * COUNT5);
main_4<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_3, d_HT_3.ref(cuco::find), d_HT_5.ref(cuco::find), d_KEY_5lineitem_u_1__l_partkey, d_aggr_rw__rw0, d_aggr_rw__rw1, d_cycles_per_warp_main_4_aggregation_5, d_cycles_per_warp_main_4_join_probe_3, lineitem_size, d_lineitem__l_partkey, d_lineitem__l_quantity, d_moved_aggr__p_partkey, d_part__p_partkey);
int64_t* cycles_per_warp_main_4_join_probe_3 = (int64_t*)malloc(sizeof(int64_t) * main_4_join_probe_3_cpw_size);
hipMemcpy(cycles_per_warp_main_4_join_probe_3, d_cycles_per_warp_main_4_join_probe_3, sizeof(int64_t) * main_4_join_probe_3_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_4_join_probe_3 ";
for (auto i=0ull; i < main_4_join_probe_3_cpw_size; i++) std::cout << cycles_per_warp_main_4_join_probe_3[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_4_aggregation_5 = (int64_t*)malloc(sizeof(int64_t) * main_4_aggregation_5_cpw_size);
hipMemcpy(cycles_per_warp_main_4_aggregation_5, d_cycles_per_warp_main_4_aggregation_5, sizeof(int64_t) * main_4_aggregation_5_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_4_aggregation_5 ";
for (auto i=0ull; i < main_4_aggregation_5_cpw_size; i++) std::cout << cycles_per_warp_main_4_aggregation_5[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_7_map_6;
auto main_7_map_6_cpw_size = std::ceil((float)COUNT5/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_map_6, sizeof(int64_t) * main_7_map_6_cpw_size);
hipMemset(d_cycles_per_warp_main_7_map_6, -1, sizeof(int64_t) * main_7_map_6_cpw_size);
int64_t* d_cycles_per_warp_main_7_map_8;
auto main_7_map_8_cpw_size = std::ceil((float)COUNT5/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_map_8, sizeof(int64_t) * main_7_map_8_cpw_size);
hipMemset(d_cycles_per_warp_main_7_map_8, -1, sizeof(int64_t) * main_7_map_8_cpw_size);
int64_t* d_cycles_per_warp_main_7_selection_9;
auto main_7_selection_9_cpw_size = std::ceil((float)COUNT5/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_selection_9, sizeof(int64_t) * main_7_selection_9_cpw_size);
hipMemset(d_cycles_per_warp_main_7_selection_9, -1, sizeof(int64_t) * main_7_selection_9_cpw_size);
//Materialize count
uint64_t* d_COUNT10;
hipMalloc(&d_COUNT10, sizeof(uint64_t));
hipMemset(d_COUNT10, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)COUNT5/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT10, COUNT5, d_moved_aggr__p_partkey);
uint64_t COUNT10;
hipMemcpy(&COUNT10, d_COUNT10, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_7_join_build_10;
auto main_7_join_build_10_cpw_size = std::ceil((float)COUNT5/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_build_10, sizeof(int64_t) * main_7_join_build_10_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_build_10, -1, sizeof(int64_t) * main_7_join_build_10_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_10;
hipMalloc(&d_BUF_IDX_10, sizeof(uint64_t));
hipMemset(d_BUF_IDX_10, 0, sizeof(uint64_t));
uint64_t* d_BUF_10;
hipMalloc(&d_BUF_10, sizeof(uint64_t) * COUNT10 * 1);
auto d_HT_10 = cuco::static_map{ (int)COUNT10*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)COUNT5/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_10, d_BUF_IDX_10, COUNT5, d_HT_10.ref(cuco::insert), d_cycles_per_warp_main_7_join_build_10, d_cycles_per_warp_main_7_map_6, d_cycles_per_warp_main_7_map_8, d_cycles_per_warp_main_7_selection_9, d_moved_aggr__p_partkey);
int64_t* cycles_per_warp_main_7_map_6 = (int64_t*)malloc(sizeof(int64_t) * main_7_map_6_cpw_size);
hipMemcpy(cycles_per_warp_main_7_map_6, d_cycles_per_warp_main_7_map_6, sizeof(int64_t) * main_7_map_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_map_6 ";
for (auto i=0ull; i < main_7_map_6_cpw_size; i++) std::cout << cycles_per_warp_main_7_map_6[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_map_8 = (int64_t*)malloc(sizeof(int64_t) * main_7_map_8_cpw_size);
hipMemcpy(cycles_per_warp_main_7_map_8, d_cycles_per_warp_main_7_map_8, sizeof(int64_t) * main_7_map_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_map_8 ";
for (auto i=0ull; i < main_7_map_8_cpw_size; i++) std::cout << cycles_per_warp_main_7_map_8[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_selection_9 = (int64_t*)malloc(sizeof(int64_t) * main_7_selection_9_cpw_size);
hipMemcpy(cycles_per_warp_main_7_selection_9, d_cycles_per_warp_main_7_selection_9, sizeof(int64_t) * main_7_selection_9_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_selection_9 ";
for (auto i=0ull; i < main_7_selection_9_cpw_size; i++) std::cout << cycles_per_warp_main_7_selection_9[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_join_build_10 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_build_10_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_build_10, d_cycles_per_warp_main_7_join_build_10, sizeof(int64_t) * main_7_join_build_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_build_10 ";
for (auto i=0ull; i < main_7_join_build_10_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_build_10[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_11_join_probe_10;
auto main_11_join_probe_10_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_join_probe_10, sizeof(int64_t) * main_11_join_probe_10_cpw_size);
hipMemset(d_cycles_per_warp_main_11_join_probe_10, -1, sizeof(int64_t) * main_11_join_probe_10_cpw_size);
//Create aggregation hash table
auto d_HT_12 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_11<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_10, d_HT_10.ref(cuco::find), d_HT_12.ref(cuco::insert), d_aggr_rw__rw0, d_aggr_rw__rw1, d_lineitem__l_partkey, d_lineitem__l_quantity, lineitem_size);
size_t COUNT12 = d_HT_12.size();
thrust::device_vector<int64_t> keys_12(COUNT12), vals_12(COUNT12);
d_HT_12.retrieve_all(keys_12.begin(), vals_12.begin());
d_HT_12.clear();
int64_t* raw_keys12 = thrust::raw_pointer_cast(keys_12.data());
insertKeys<<<std::ceil((float)COUNT12/128.), 128>>>(raw_keys12, d_HT_12.ref(cuco::insert), COUNT12);
int64_t* d_cycles_per_warp_main_11_aggregation_12;
auto main_11_aggregation_12_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_aggregation_12, sizeof(int64_t) * main_11_aggregation_12_cpw_size);
hipMemset(d_cycles_per_warp_main_11_aggregation_12, -1, sizeof(int64_t) * main_11_aggregation_12_cpw_size);
//Aggregate in hashtable
DBDecimalType* d_aggr1__tmp_attr2;
hipMalloc(&d_aggr1__tmp_attr2, sizeof(DBDecimalType) * COUNT12);
hipMemset(d_aggr1__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT12);
main_11<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_10, d_HT_10.ref(cuco::find), d_HT_12.ref(cuco::find), d_aggr1__tmp_attr2, d_aggr_rw__rw0, d_aggr_rw__rw1, d_cycles_per_warp_main_11_aggregation_12, d_cycles_per_warp_main_11_join_probe_10, d_lineitem__l_extendedprice, d_lineitem__l_partkey, d_lineitem__l_quantity, lineitem_size);
int64_t* cycles_per_warp_main_11_join_probe_10 = (int64_t*)malloc(sizeof(int64_t) * main_11_join_probe_10_cpw_size);
hipMemcpy(cycles_per_warp_main_11_join_probe_10, d_cycles_per_warp_main_11_join_probe_10, sizeof(int64_t) * main_11_join_probe_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_join_probe_10 ";
for (auto i=0ull; i < main_11_join_probe_10_cpw_size; i++) std::cout << cycles_per_warp_main_11_join_probe_10[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_11_aggregation_12 = (int64_t*)malloc(sizeof(int64_t) * main_11_aggregation_12_cpw_size);
hipMemcpy(cycles_per_warp_main_11_aggregation_12, d_cycles_per_warp_main_11_aggregation_12, sizeof(int64_t) * main_11_aggregation_12_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_aggregation_12 ";
for (auto i=0ull; i < main_11_aggregation_12_cpw_size; i++) std::cout << cycles_per_warp_main_11_aggregation_12[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_14_map_13;
auto main_14_map_13_cpw_size = std::ceil((float)COUNT12/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_14_map_13, sizeof(int64_t) * main_14_map_13_cpw_size);
hipMemset(d_cycles_per_warp_main_14_map_13, -1, sizeof(int64_t) * main_14_map_13_cpw_size);
//Materialize count
uint64_t* d_COUNT15;
hipMalloc(&d_COUNT15, sizeof(uint64_t));
hipMemset(d_COUNT15, 0, sizeof(uint64_t));
count_14<<<std::ceil((float)COUNT12/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT12, d_COUNT15);
uint64_t COUNT15;
hipMemcpy(&COUNT15, d_COUNT15, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_14_materialize_15;
auto main_14_materialize_15_cpw_size = std::ceil((float)COUNT12/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_14_materialize_15, sizeof(int64_t) * main_14_materialize_15_cpw_size);
hipMemset(d_cycles_per_warp_main_14_materialize_15, -1, sizeof(int64_t) * main_14_materialize_15_cpw_size);
//Materialize buffers
uint64_t* d_MAT_IDX15;
hipMalloc(&d_MAT_IDX15, sizeof(uint64_t));
hipMemset(d_MAT_IDX15, 0, sizeof(uint64_t));
auto MAT15map1__tmp_attr3 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT15);
DBDecimalType* d_MAT15map1__tmp_attr3;
hipMalloc(&d_MAT15map1__tmp_attr3, sizeof(DBDecimalType) * COUNT15);
main_14<<<std::ceil((float)COUNT12/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT12, d_MAT15map1__tmp_attr3, d_MAT_IDX15, d_aggr1__tmp_attr2, d_cycles_per_warp_main_14_map_13, d_cycles_per_warp_main_14_materialize_15);
hipMemcpy(MAT15map1__tmp_attr3, d_MAT15map1__tmp_attr3, sizeof(DBDecimalType) * COUNT15, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_14_map_13 = (int64_t*)malloc(sizeof(int64_t) * main_14_map_13_cpw_size);
hipMemcpy(cycles_per_warp_main_14_map_13, d_cycles_per_warp_main_14_map_13, sizeof(int64_t) * main_14_map_13_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_14_map_13 ";
for (auto i=0ull; i < main_14_map_13_cpw_size; i++) std::cout << cycles_per_warp_main_14_map_13[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_14_materialize_15 = (int64_t*)malloc(sizeof(int64_t) * main_14_materialize_15_cpw_size);
hipMemcpy(cycles_per_warp_main_14_materialize_15, d_cycles_per_warp_main_14_materialize_15, sizeof(int64_t) * main_14_materialize_15_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_14_materialize_15 ";
for (auto i=0ull; i < main_14_materialize_15_cpw_size; i++) std::cout << cycles_per_warp_main_14_materialize_15[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
      size_t aux_mem = usedGpuMem() - used_mem;
      std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_3);
hipFree(d_BUF_IDX_3);
hipFree(d_COUNT3);
hipFree(d_KEY_5lineitem_u_1__l_partkey);
hipFree(d_aggr_rw__rw0);
hipFree(d_aggr_rw__rw1);
hipFree(d_moved_aggr__p_partkey);
hipFree(d_BUF_10);
hipFree(d_BUF_IDX_10);
hipFree(d_COUNT10);
hipFree(d_aggr1__tmp_attr2);
hipFree(d_COUNT15);
hipFree(d_MAT15map1__tmp_attr3);
hipFree(d_MAT_IDX15);
free(MAT15map1__tmp_attr3);
}