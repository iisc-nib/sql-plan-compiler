#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_INSERT HT_2, int64_t* cycles_per_warp_main_1_join_build_2, int64_t* cycles_per_warp_main_1_selection_0, DBI32Type* part__p_partkey, DBStringType* part__p_type, size_t part_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_part__p_type[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_type[ITEM] = part__p_type[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_part__p_type[ITEM], "ECONOMY ANODIZED STEEL", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_2.insert(cuco::pair{KEY_2[ITEM], ITEM*TB + tid});
BUF_2[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_build_2[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_INSERT HT_6, int64_t* cycles_per_warp_main_5_join_build_6, int64_t* cycles_per_warp_main_5_selection_4, DBStringType* region__r_name, DBI32Type* region__r_regionkey, size_t region_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_region__r_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
reg_region__r_name[ITEM] = region__r_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_region__r_name[ITEM], "AMERICA", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_selection_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_region__r_regionkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
reg_region__r_regionkey[ITEM] = region__r_regionkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_region__r_regionkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_6.insert(cuco::pair{KEY_6[ITEM], ITEM*TB + tid});
BUF_6[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_join_build_6[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT>
__global__ void main_9(uint64_t* BUF_10, uint64_t* BUF_IDX_10, HASHTABLE_INSERT HT_10, int64_t* cycles_per_warp_main_9_join_build_10, int64_t* cycles_per_warp_main_9_selection_8, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_orders__o_orderdate[ITEM], 9131, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate[ITEM], 9861, Predicate::lte);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_9_selection_8[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_orders__o_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_10.insert(cuco::pair{KEY_10[ITEM], ITEM*TB + tid});
BUF_10[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_9_join_build_10[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT>
__global__ void main_12(uint64_t* BUF_11, uint64_t* BUF_IDX_11, HASHTABLE_INSERT HT_11, DBI32Type* customer__c_custkey, size_t customer_size, int64_t* cycles_per_warp_main_12_join_build_11) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_11[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_11[ITEM] = 0;
KEY_11[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_11.insert(cuco::pair{KEY_11[ITEM], ITEM*TB + tid});
BUF_11[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_12_join_build_11[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_7(uint64_t* BUF_13, uint64_t* BUF_6, uint64_t* BUF_IDX_13, HASHTABLE_INSERT HT_13, HASHTABLE_PROBE HT_6, int64_t* cycles_per_warp_main_7_join_build_13, int64_t* cycles_per_warp_main_7_join_probe_6, DBI32Type* n1___n_nationkey, DBI32Type* n1___n_regionkey, size_t nation_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_n1___n_regionkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n1___n_regionkey[ITEM] = n1___n_regionkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_n1___n_regionkey[ITEM];
}
//Probe Hash table
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_probe_6[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_13[ITEMS_PER_THREAD];
DBI32Type reg_n1___n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n1___n_nationkey[ITEM] = n1___n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_13[ITEM] = 0;
KEY_13[ITEM] |= reg_n1___n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_13.insert(cuco::pair{KEY_13[ITEM], ITEM*TB + tid});
BUF_13[(ITEM*TB + tid) * 2 + 0] = BUF_6[slot_second6[ITEM] * 1 + 0];
BUF_13[(ITEM*TB + tid) * 2 + 1] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_build_13[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT>
__global__ void main_15(uint64_t* BUF_14, uint64_t* BUF_IDX_14, HASHTABLE_INSERT HT_14, int64_t* cycles_per_warp_main_15_join_build_14, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_14[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_14[ITEM] = 0;
KEY_14[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_14.insert(cuco::pair{KEY_14[ITEM], ITEM*TB + tid});
BUF_14[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_15_join_build_14[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT>
__global__ void main_17(uint64_t* BUF_16, uint64_t* BUF_IDX_16, HASHTABLE_INSERT HT_16, int64_t* cycles_per_warp_main_17_join_build_16, DBI32Type* n2___n_nationkey, size_t nation_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_16[ITEMS_PER_THREAD];
DBI32Type reg_n2___n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n2___n_nationkey[ITEM] = n2___n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_16[ITEM] = 0;
KEY_16[ITEM] |= reg_n2___n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_16.insert(cuco::pair{KEY_16[ITEM], ITEM*TB + tid});
BUF_16[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_join_build_16[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_3(uint64_t* BUF_10, uint64_t* BUF_11, uint64_t* BUF_13, uint64_t* BUF_14, uint64_t* BUF_16, uint64_t* BUF_2, HASHTABLE_PROBE HT_10, HASHTABLE_PROBE HT_11, HASHTABLE_PROBE HT_13, HASHTABLE_PROBE HT_14, HASHTABLE_PROBE HT_16, HASHTABLE_PROBE HT_2, HASHTABLE_FIND HT_21, DBI64Type* KEY_21map0__tmp_attr0, int* SLOT_COUNT_21, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, DBI32Type* customer__c_nationkey, int64_t* cycles_per_warp_main_3_aggregation_21, int64_t* cycles_per_warp_main_3_join_probe_10, int64_t* cycles_per_warp_main_3_join_probe_11, int64_t* cycles_per_warp_main_3_join_probe_13, int64_t* cycles_per_warp_main_3_join_probe_14, int64_t* cycles_per_warp_main_3_join_probe_16, int64_t* cycles_per_warp_main_3_join_probe_2, int64_t* cycles_per_warp_main_3_map_18, int64_t* cycles_per_warp_main_3_map_19, int64_t* cycles_per_warp_main_3_map_20, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n2___n_name, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* supplier__s_nationkey) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
//Probe Hash table
int64_t slot_second2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_2 = HT_2.find(KEY_2[ITEM]);
if (SLOT_2 == HT_2.end()) {selection_flags[ITEM] = 0; continue;}
slot_second2[ITEM] = SLOT_2->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_probe_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
//Probe Hash table
int64_t slot_second10[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_10 = HT_10.find(KEY_10[ITEM]);
if (SLOT_10 == HT_10.end()) {selection_flags[ITEM] = 0; continue;}
slot_second10[ITEM] = SLOT_10->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_probe_10[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_11[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_custkey[ITEM] = orders__o_custkey[BUF_10[slot_second10[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_11[ITEM] = 0;
KEY_11[ITEM] |= reg_orders__o_custkey[ITEM];
}
//Probe Hash table
int64_t slot_second11[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_11 = HT_11.find(KEY_11[ITEM]);
if (SLOT_11 == HT_11.end()) {selection_flags[ITEM] = 0; continue;}
slot_second11[ITEM] = SLOT_11->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_probe_11[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_13[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[BUF_11[slot_second11[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_13[ITEM] = 0;
KEY_13[ITEM] |= reg_customer__c_nationkey[ITEM];
}
//Probe Hash table
int64_t slot_second13[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_13 = HT_13.find(KEY_13[ITEM]);
if (SLOT_13 == HT_13.end()) {selection_flags[ITEM] = 0; continue;}
slot_second13[ITEM] = SLOT_13->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_probe_13[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_14[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_suppkey[ITEM] = lineitem__l_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_14[ITEM] = 0;
KEY_14[ITEM] |= reg_lineitem__l_suppkey[ITEM];
}
//Probe Hash table
int64_t slot_second14[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_14 = HT_14.find(KEY_14[ITEM]);
if (SLOT_14 == HT_14.end()) {selection_flags[ITEM] = 0; continue;}
slot_second14[ITEM] = SLOT_14->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_probe_14[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_16[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_supplier__s_nationkey[ITEM] = supplier__s_nationkey[BUF_14[slot_second14[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_16[ITEM] = 0;
KEY_16[ITEM] |= reg_supplier__s_nationkey[ITEM];
}
//Probe Hash table
int64_t slot_second16[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_16 = HT_16.find(KEY_16[ITEM]);
if (SLOT_16 == HT_16.end()) {selection_flags[ITEM] = 0; continue;}
slot_second16[ITEM] = SLOT_16->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_probe_16[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_map_18[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[BUF_10[slot_second10[ITEM] * 1 + 0]];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_map_19[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_map_20[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_21[ITEMS_PER_THREAD];
DBI64Type reg_map0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr0[ITEM] = ExtractFromDate("year", reg_orders__o_orderdate[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_21[ITEM] = 0;
KEY_21[ITEM] |= (DBI32Type)reg_map0__tmp_attr0[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]));
}
DBStringType reg_n2___n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_n2___n_name[ITEM] = n2___n_name[BUF_16[slot_second16[ITEM] * 1 + 0]];
}
DBDecimalType reg_map1__tmp_attr3[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map1__tmp_attr3[ITEM] = ((evaluatePredicate(reg_n2___n_name[ITEM], "BRAZIL", Predicate::eq))) ? ((reg_map0__tmp_attr1[ITEM])) : ((0.0));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_21 = get_aggregation_slot(KEY_21[ITEM], HT_21, SLOT_COUNT_21);
aggregate_sum(&aggr0__tmp_attr4[buf_idx_21], reg_map0__tmp_attr1[ITEM]);
aggregate_sum(&aggr0__tmp_attr2[buf_idx_21], reg_map1__tmp_attr3[ITEM]);
KEY_21map0__tmp_attr0[buf_idx_21] = reg_map0__tmp_attr0[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_aggregation_21[blockIdx.x] = cycles_per_warp;}
}
__global__ void main_23(size_t COUNT21, DBI64Type* MAT24map0__tmp_attr0, DBDecimalType* MAT24map2__tmp_attr5, uint64_t* MAT_IDX24, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, int64_t* cycles_per_warp_main_23_map_22, int64_t* cycles_per_warp_main_23_materialize_24, DBI64Type* map0__tmp_attr0) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_23_map_22[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBI64Type reg_map0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT21); ++ITEM) {
reg_map0__tmp_attr0[ITEM] = map0__tmp_attr0[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT21); ++ITEM) {
reg_aggr0__tmp_attr4[ITEM] = aggr0__tmp_attr4[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT21); ++ITEM) {
reg_aggr0__tmp_attr2[ITEM] = aggr0__tmp_attr2[ITEM*TB + tid];
}
DBDecimalType reg_map2__tmp_attr5[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT21); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map2__tmp_attr5[ITEM] = (reg_aggr0__tmp_attr2[ITEM]) / (reg_aggr0__tmp_attr4[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT21); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx24 = atomicAdd((int*)MAT_IDX24, 1);
MAT24map0__tmp_attr0[mat_idx24] = reg_map0__tmp_attr0[ITEM];
MAT24map2__tmp_attr5[mat_idx24] = reg_map2__tmp_attr5[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_23_materialize_24[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_join_build_2;
auto main_1_join_build_2_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_build_2, sizeof(int64_t) * main_1_join_build_2_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_build_2, -1, sizeof(int64_t) * main_1_join_build_2_cpw_size);
size_t COUNT2 = part_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 1);
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)part_size/(float)TILE_SIZE), TB>>>(d_BUF_2, d_BUF_IDX_2, d_HT_2.ref(cuco::insert), d_cycles_per_warp_main_1_join_build_2, d_cycles_per_warp_main_1_selection_0, d_part__p_partkey, d_part__p_type, part_size);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_build_2 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_build_2_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_build_2, d_cycles_per_warp_main_1_join_build_2, sizeof(int64_t) * main_1_join_build_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_build_2 ";
for (auto i=0ull; i < main_1_join_build_2_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_build_2[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_5_selection_4;
auto main_5_selection_4_cpw_size = std::ceil((float)region_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_selection_4, sizeof(int64_t) * main_5_selection_4_cpw_size);
hipMemset(d_cycles_per_warp_main_5_selection_4, -1, sizeof(int64_t) * main_5_selection_4_cpw_size);
int64_t* d_cycles_per_warp_main_5_join_build_6;
auto main_5_join_build_6_cpw_size = std::ceil((float)region_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_join_build_6, sizeof(int64_t) * main_5_join_build_6_cpw_size);
hipMemset(d_cycles_per_warp_main_5_join_build_6, -1, sizeof(int64_t) * main_5_join_build_6_cpw_size);
size_t COUNT6 = region_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 1);
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)region_size/(float)TILE_SIZE), TB>>>(d_BUF_6, d_BUF_IDX_6, d_HT_6.ref(cuco::insert), d_cycles_per_warp_main_5_join_build_6, d_cycles_per_warp_main_5_selection_4, d_region__r_name, d_region__r_regionkey, region_size);
int64_t* cycles_per_warp_main_5_selection_4 = (int64_t*)malloc(sizeof(int64_t) * main_5_selection_4_cpw_size);
hipMemcpy(cycles_per_warp_main_5_selection_4, d_cycles_per_warp_main_5_selection_4, sizeof(int64_t) * main_5_selection_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_selection_4 ";
for (auto i=0ull; i < main_5_selection_4_cpw_size; i++) std::cout << cycles_per_warp_main_5_selection_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_join_build_6 = (int64_t*)malloc(sizeof(int64_t) * main_5_join_build_6_cpw_size);
hipMemcpy(cycles_per_warp_main_5_join_build_6, d_cycles_per_warp_main_5_join_build_6, sizeof(int64_t) * main_5_join_build_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_join_build_6 ";
for (auto i=0ull; i < main_5_join_build_6_cpw_size; i++) std::cout << cycles_per_warp_main_5_join_build_6[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_9_selection_8;
auto main_9_selection_8_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_9_selection_8, sizeof(int64_t) * main_9_selection_8_cpw_size);
hipMemset(d_cycles_per_warp_main_9_selection_8, -1, sizeof(int64_t) * main_9_selection_8_cpw_size);
int64_t* d_cycles_per_warp_main_9_join_build_10;
auto main_9_join_build_10_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_9_join_build_10, sizeof(int64_t) * main_9_join_build_10_cpw_size);
hipMemset(d_cycles_per_warp_main_9_join_build_10, -1, sizeof(int64_t) * main_9_join_build_10_cpw_size);
size_t COUNT10 = orders_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_10;
hipMalloc(&d_BUF_IDX_10, sizeof(uint64_t));
hipMemset(d_BUF_IDX_10, 0, sizeof(uint64_t));
uint64_t* d_BUF_10;
hipMalloc(&d_BUF_10, sizeof(uint64_t) * COUNT10 * 1);
auto d_HT_10 = cuco::static_map{ (int)COUNT10*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_9<<<std::ceil((float)orders_size/(float)TILE_SIZE), TB>>>(d_BUF_10, d_BUF_IDX_10, d_HT_10.ref(cuco::insert), d_cycles_per_warp_main_9_join_build_10, d_cycles_per_warp_main_9_selection_8, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
int64_t* cycles_per_warp_main_9_selection_8 = (int64_t*)malloc(sizeof(int64_t) * main_9_selection_8_cpw_size);
hipMemcpy(cycles_per_warp_main_9_selection_8, d_cycles_per_warp_main_9_selection_8, sizeof(int64_t) * main_9_selection_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_9_selection_8 ";
for (auto i=0ull; i < main_9_selection_8_cpw_size; i++) std::cout << cycles_per_warp_main_9_selection_8[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_9_join_build_10 = (int64_t*)malloc(sizeof(int64_t) * main_9_join_build_10_cpw_size);
hipMemcpy(cycles_per_warp_main_9_join_build_10, d_cycles_per_warp_main_9_join_build_10, sizeof(int64_t) * main_9_join_build_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_9_join_build_10 ";
for (auto i=0ull; i < main_9_join_build_10_cpw_size; i++) std::cout << cycles_per_warp_main_9_join_build_10[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_12_join_build_11;
auto main_12_join_build_11_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_12_join_build_11, sizeof(int64_t) * main_12_join_build_11_cpw_size);
hipMemset(d_cycles_per_warp_main_12_join_build_11, -1, sizeof(int64_t) * main_12_join_build_11_cpw_size);
size_t COUNT11 = customer_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_11;
hipMalloc(&d_BUF_IDX_11, sizeof(uint64_t));
hipMemset(d_BUF_IDX_11, 0, sizeof(uint64_t));
uint64_t* d_BUF_11;
hipMalloc(&d_BUF_11, sizeof(uint64_t) * COUNT11 * 1);
auto d_HT_11 = cuco::static_map{ (int)COUNT11*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_12<<<std::ceil((float)customer_size/(float)TILE_SIZE), TB>>>(d_BUF_11, d_BUF_IDX_11, d_HT_11.ref(cuco::insert), d_customer__c_custkey, customer_size, d_cycles_per_warp_main_12_join_build_11);
int64_t* cycles_per_warp_main_12_join_build_11 = (int64_t*)malloc(sizeof(int64_t) * main_12_join_build_11_cpw_size);
hipMemcpy(cycles_per_warp_main_12_join_build_11, d_cycles_per_warp_main_12_join_build_11, sizeof(int64_t) * main_12_join_build_11_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_12_join_build_11 ";
for (auto i=0ull; i < main_12_join_build_11_cpw_size; i++) std::cout << cycles_per_warp_main_12_join_build_11[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_7_join_probe_6;
auto main_7_join_probe_6_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_probe_6, sizeof(int64_t) * main_7_join_probe_6_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_probe_6, -1, sizeof(int64_t) * main_7_join_probe_6_cpw_size);
int64_t* d_cycles_per_warp_main_7_join_build_13;
auto main_7_join_build_13_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_build_13, sizeof(int64_t) * main_7_join_build_13_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_build_13, -1, sizeof(int64_t) * main_7_join_build_13_cpw_size);
size_t COUNT13 = nation_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_13;
hipMalloc(&d_BUF_IDX_13, sizeof(uint64_t));
hipMemset(d_BUF_IDX_13, 0, sizeof(uint64_t));
uint64_t* d_BUF_13;
hipMalloc(&d_BUF_13, sizeof(uint64_t) * COUNT13 * 2);
auto d_HT_13 = cuco::static_map{ (int)COUNT13*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)nation_size/(float)TILE_SIZE), TB>>>(d_BUF_13, d_BUF_6, d_BUF_IDX_13, d_HT_13.ref(cuco::insert), d_HT_6.ref(cuco::find), d_cycles_per_warp_main_7_join_build_13, d_cycles_per_warp_main_7_join_probe_6, d_nation__n_nationkey, d_nation__n_regionkey, nation_size);
int64_t* cycles_per_warp_main_7_join_probe_6 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_probe_6_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_probe_6, d_cycles_per_warp_main_7_join_probe_6, sizeof(int64_t) * main_7_join_probe_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_probe_6 ";
for (auto i=0ull; i < main_7_join_probe_6_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_probe_6[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_join_build_13 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_build_13_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_build_13, d_cycles_per_warp_main_7_join_build_13, sizeof(int64_t) * main_7_join_build_13_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_build_13 ";
for (auto i=0ull; i < main_7_join_build_13_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_build_13[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_15_join_build_14;
auto main_15_join_build_14_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_15_join_build_14, sizeof(int64_t) * main_15_join_build_14_cpw_size);
hipMemset(d_cycles_per_warp_main_15_join_build_14, -1, sizeof(int64_t) * main_15_join_build_14_cpw_size);
size_t COUNT14 = supplier_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_14;
hipMalloc(&d_BUF_IDX_14, sizeof(uint64_t));
hipMemset(d_BUF_IDX_14, 0, sizeof(uint64_t));
uint64_t* d_BUF_14;
hipMalloc(&d_BUF_14, sizeof(uint64_t) * COUNT14 * 1);
auto d_HT_14 = cuco::static_map{ (int)COUNT14*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_15<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TB>>>(d_BUF_14, d_BUF_IDX_14, d_HT_14.ref(cuco::insert), d_cycles_per_warp_main_15_join_build_14, d_supplier__s_suppkey, supplier_size);
int64_t* cycles_per_warp_main_15_join_build_14 = (int64_t*)malloc(sizeof(int64_t) * main_15_join_build_14_cpw_size);
hipMemcpy(cycles_per_warp_main_15_join_build_14, d_cycles_per_warp_main_15_join_build_14, sizeof(int64_t) * main_15_join_build_14_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_15_join_build_14 ";
for (auto i=0ull; i < main_15_join_build_14_cpw_size; i++) std::cout << cycles_per_warp_main_15_join_build_14[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_17_join_build_16;
auto main_17_join_build_16_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_join_build_16, sizeof(int64_t) * main_17_join_build_16_cpw_size);
hipMemset(d_cycles_per_warp_main_17_join_build_16, -1, sizeof(int64_t) * main_17_join_build_16_cpw_size);
size_t COUNT16 = nation_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_16;
hipMalloc(&d_BUF_IDX_16, sizeof(uint64_t));
hipMemset(d_BUF_IDX_16, 0, sizeof(uint64_t));
uint64_t* d_BUF_16;
hipMalloc(&d_BUF_16, sizeof(uint64_t) * COUNT16 * 1);
auto d_HT_16 = cuco::static_map{ (int)COUNT16*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_17<<<std::ceil((float)nation_size/(float)TILE_SIZE), TB>>>(d_BUF_16, d_BUF_IDX_16, d_HT_16.ref(cuco::insert), d_cycles_per_warp_main_17_join_build_16, d_nation__n_nationkey, nation_size);
int64_t* cycles_per_warp_main_17_join_build_16 = (int64_t*)malloc(sizeof(int64_t) * main_17_join_build_16_cpw_size);
hipMemcpy(cycles_per_warp_main_17_join_build_16, d_cycles_per_warp_main_17_join_build_16, sizeof(int64_t) * main_17_join_build_16_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_join_build_16 ";
for (auto i=0ull; i < main_17_join_build_16_cpw_size; i++) std::cout << cycles_per_warp_main_17_join_build_16[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_3_join_probe_2;
auto main_3_join_probe_2_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_probe_2, sizeof(int64_t) * main_3_join_probe_2_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_probe_2, -1, sizeof(int64_t) * main_3_join_probe_2_cpw_size);
int64_t* d_cycles_per_warp_main_3_join_probe_10;
auto main_3_join_probe_10_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_probe_10, sizeof(int64_t) * main_3_join_probe_10_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_probe_10, -1, sizeof(int64_t) * main_3_join_probe_10_cpw_size);
int64_t* d_cycles_per_warp_main_3_join_probe_11;
auto main_3_join_probe_11_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_probe_11, sizeof(int64_t) * main_3_join_probe_11_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_probe_11, -1, sizeof(int64_t) * main_3_join_probe_11_cpw_size);
int64_t* d_cycles_per_warp_main_3_join_probe_13;
auto main_3_join_probe_13_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_probe_13, sizeof(int64_t) * main_3_join_probe_13_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_probe_13, -1, sizeof(int64_t) * main_3_join_probe_13_cpw_size);
int64_t* d_cycles_per_warp_main_3_join_probe_14;
auto main_3_join_probe_14_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_probe_14, sizeof(int64_t) * main_3_join_probe_14_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_probe_14, -1, sizeof(int64_t) * main_3_join_probe_14_cpw_size);
int64_t* d_cycles_per_warp_main_3_join_probe_16;
auto main_3_join_probe_16_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_probe_16, sizeof(int64_t) * main_3_join_probe_16_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_probe_16, -1, sizeof(int64_t) * main_3_join_probe_16_cpw_size);
int64_t* d_cycles_per_warp_main_3_map_18;
auto main_3_map_18_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_map_18, sizeof(int64_t) * main_3_map_18_cpw_size);
hipMemset(d_cycles_per_warp_main_3_map_18, -1, sizeof(int64_t) * main_3_map_18_cpw_size);
int64_t* d_cycles_per_warp_main_3_map_19;
auto main_3_map_19_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_map_19, sizeof(int64_t) * main_3_map_19_cpw_size);
hipMemset(d_cycles_per_warp_main_3_map_19, -1, sizeof(int64_t) * main_3_map_19_cpw_size);
int64_t* d_cycles_per_warp_main_3_map_20;
auto main_3_map_20_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_map_20, sizeof(int64_t) * main_3_map_20_cpw_size);
hipMemset(d_cycles_per_warp_main_3_map_20, -1, sizeof(int64_t) * main_3_map_20_cpw_size);
int64_t* d_cycles_per_warp_main_3_aggregation_21;
auto main_3_aggregation_21_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_aggregation_21, sizeof(int64_t) * main_3_aggregation_21_cpw_size);
hipMemset(d_cycles_per_warp_main_3_aggregation_21, -1, sizeof(int64_t) * main_3_aggregation_21_cpw_size);
size_t COUNT21 = 5117;
auto d_HT_21 = cuco::static_map{ (int)5117*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_21;
hipMalloc(&d_SLOT_COUNT_21, sizeof(int));
hipMemset(d_SLOT_COUNT_21, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr4;
hipMalloc(&d_aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT21);
hipMemset(d_aggr0__tmp_attr4, 0, sizeof(DBDecimalType) * COUNT21);
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT21);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT21);
DBI64Type* d_KEY_21map0__tmp_attr0;
hipMalloc(&d_KEY_21map0__tmp_attr0, sizeof(DBI64Type) * COUNT21);
hipMemset(d_KEY_21map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT21);
main_3<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TB>>>(d_BUF_10, d_BUF_11, d_BUF_13, d_BUF_14, d_BUF_16, d_BUF_2, d_HT_10.ref(cuco::find), d_HT_11.ref(cuco::find), d_HT_13.ref(cuco::find), d_HT_14.ref(cuco::find), d_HT_16.ref(cuco::find), d_HT_2.ref(cuco::find), d_HT_21.ref(cuco::insert_and_find), d_KEY_21map0__tmp_attr0, d_SLOT_COUNT_21, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_customer__c_nationkey, d_cycles_per_warp_main_3_aggregation_21, d_cycles_per_warp_main_3_join_probe_10, d_cycles_per_warp_main_3_join_probe_11, d_cycles_per_warp_main_3_join_probe_13, d_cycles_per_warp_main_3_join_probe_14, d_cycles_per_warp_main_3_join_probe_16, d_cycles_per_warp_main_3_join_probe_2, d_cycles_per_warp_main_3_map_18, d_cycles_per_warp_main_3_map_19, d_cycles_per_warp_main_3_map_20, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_orders__o_custkey, d_orders__o_orderdate, d_supplier__s_nationkey);
int64_t* cycles_per_warp_main_3_join_probe_2 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_probe_2_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_probe_2, d_cycles_per_warp_main_3_join_probe_2, sizeof(int64_t) * main_3_join_probe_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_probe_2 ";
for (auto i=0ull; i < main_3_join_probe_2_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_probe_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_join_probe_10 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_probe_10_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_probe_10, d_cycles_per_warp_main_3_join_probe_10, sizeof(int64_t) * main_3_join_probe_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_probe_10 ";
for (auto i=0ull; i < main_3_join_probe_10_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_probe_10[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_join_probe_11 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_probe_11_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_probe_11, d_cycles_per_warp_main_3_join_probe_11, sizeof(int64_t) * main_3_join_probe_11_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_probe_11 ";
for (auto i=0ull; i < main_3_join_probe_11_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_probe_11[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_join_probe_13 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_probe_13_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_probe_13, d_cycles_per_warp_main_3_join_probe_13, sizeof(int64_t) * main_3_join_probe_13_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_probe_13 ";
for (auto i=0ull; i < main_3_join_probe_13_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_probe_13[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_join_probe_14 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_probe_14_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_probe_14, d_cycles_per_warp_main_3_join_probe_14, sizeof(int64_t) * main_3_join_probe_14_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_probe_14 ";
for (auto i=0ull; i < main_3_join_probe_14_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_probe_14[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_join_probe_16 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_probe_16_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_probe_16, d_cycles_per_warp_main_3_join_probe_16, sizeof(int64_t) * main_3_join_probe_16_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_probe_16 ";
for (auto i=0ull; i < main_3_join_probe_16_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_probe_16[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_map_18 = (int64_t*)malloc(sizeof(int64_t) * main_3_map_18_cpw_size);
hipMemcpy(cycles_per_warp_main_3_map_18, d_cycles_per_warp_main_3_map_18, sizeof(int64_t) * main_3_map_18_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_map_18 ";
for (auto i=0ull; i < main_3_map_18_cpw_size; i++) std::cout << cycles_per_warp_main_3_map_18[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_map_19 = (int64_t*)malloc(sizeof(int64_t) * main_3_map_19_cpw_size);
hipMemcpy(cycles_per_warp_main_3_map_19, d_cycles_per_warp_main_3_map_19, sizeof(int64_t) * main_3_map_19_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_map_19 ";
for (auto i=0ull; i < main_3_map_19_cpw_size; i++) std::cout << cycles_per_warp_main_3_map_19[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_map_20 = (int64_t*)malloc(sizeof(int64_t) * main_3_map_20_cpw_size);
hipMemcpy(cycles_per_warp_main_3_map_20, d_cycles_per_warp_main_3_map_20, sizeof(int64_t) * main_3_map_20_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_map_20 ";
for (auto i=0ull; i < main_3_map_20_cpw_size; i++) std::cout << cycles_per_warp_main_3_map_20[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_aggregation_21 = (int64_t*)malloc(sizeof(int64_t) * main_3_aggregation_21_cpw_size);
hipMemcpy(cycles_per_warp_main_3_aggregation_21, d_cycles_per_warp_main_3_aggregation_21, sizeof(int64_t) * main_3_aggregation_21_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_aggregation_21 ";
for (auto i=0ull; i < main_3_aggregation_21_cpw_size; i++) std::cout << cycles_per_warp_main_3_aggregation_21[i] << " ";
std::cout << std::endl;
COUNT21 = d_HT_21.size();
int64_t* d_cycles_per_warp_main_23_map_22;
auto main_23_map_22_cpw_size = std::ceil((float)COUNT21/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_23_map_22, sizeof(int64_t) * main_23_map_22_cpw_size);
hipMemset(d_cycles_per_warp_main_23_map_22, -1, sizeof(int64_t) * main_23_map_22_cpw_size);
int64_t* d_cycles_per_warp_main_23_materialize_24;
auto main_23_materialize_24_cpw_size = std::ceil((float)COUNT21/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_23_materialize_24, sizeof(int64_t) * main_23_materialize_24_cpw_size);
hipMemset(d_cycles_per_warp_main_23_materialize_24, -1, sizeof(int64_t) * main_23_materialize_24_cpw_size);
size_t COUNT24 = COUNT21;
//Materialize buffers
uint64_t* d_MAT_IDX24;
hipMalloc(&d_MAT_IDX24, sizeof(uint64_t));
hipMemset(d_MAT_IDX24, 0, sizeof(uint64_t));
auto MAT24map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT24);
DBI64Type* d_MAT24map0__tmp_attr0;
hipMalloc(&d_MAT24map0__tmp_attr0, sizeof(DBI64Type) * COUNT24);
auto MAT24map2__tmp_attr5 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT24);
DBDecimalType* d_MAT24map2__tmp_attr5;
hipMalloc(&d_MAT24map2__tmp_attr5, sizeof(DBDecimalType) * COUNT24);
main_23<<<std::ceil((float)COUNT21/(float)TILE_SIZE), TB>>>(COUNT21, d_MAT24map0__tmp_attr0, d_MAT24map2__tmp_attr5, d_MAT_IDX24, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_cycles_per_warp_main_23_map_22, d_cycles_per_warp_main_23_materialize_24, d_KEY_21map0__tmp_attr0);
uint64_t MATCOUNT_24 = 0;
hipMemcpy(&MATCOUNT_24, d_MAT_IDX24, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT24map0__tmp_attr0, d_MAT24map0__tmp_attr0, sizeof(DBI64Type) * COUNT24, hipMemcpyDeviceToHost);
hipMemcpy(MAT24map2__tmp_attr5, d_MAT24map2__tmp_attr5, sizeof(DBDecimalType) * COUNT24, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_23_map_22 = (int64_t*)malloc(sizeof(int64_t) * main_23_map_22_cpw_size);
hipMemcpy(cycles_per_warp_main_23_map_22, d_cycles_per_warp_main_23_map_22, sizeof(int64_t) * main_23_map_22_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_23_map_22 ";
for (auto i=0ull; i < main_23_map_22_cpw_size; i++) std::cout << cycles_per_warp_main_23_map_22[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_23_materialize_24 = (int64_t*)malloc(sizeof(int64_t) * main_23_materialize_24_cpw_size);
hipMemcpy(cycles_per_warp_main_23_materialize_24, d_cycles_per_warp_main_23_materialize_24, sizeof(int64_t) * main_23_materialize_24_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_23_materialize_24 ";
for (auto i=0ull; i < main_23_materialize_24_cpw_size; i++) std::cout << cycles_per_warp_main_23_materialize_24[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_BUF_10);
hipFree(d_BUF_IDX_10);
hipFree(d_BUF_11);
hipFree(d_BUF_IDX_11);
hipFree(d_BUF_13);
hipFree(d_BUF_IDX_13);
hipFree(d_BUF_14);
hipFree(d_BUF_IDX_14);
hipFree(d_BUF_16);
hipFree(d_BUF_IDX_16);
hipFree(d_KEY_21map0__tmp_attr0);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_aggr0__tmp_attr4);
hipFree(d_MAT24map0__tmp_attr0);
hipFree(d_MAT24map2__tmp_attr5);
hipFree(d_MAT_IDX24);
free(MAT24map0__tmp_attr0);
free(MAT24map2__tmp_attr5);
}