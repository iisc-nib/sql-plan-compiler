#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_61641c7a9ca0(uint64_t* COUNT61641c7bca00, DBStringType* customer__c_mktsegment, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_mktsegment = customer__c_mktsegment[tid];
if (!(evaluatePredicate(reg_customer__c_mktsegment, "BUILDING", Predicate::eq))) return;
//Materialize count
atomicAdd((int*)COUNT61641c7bca00, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_61641c7a9ca0(uint64_t* BUF_61641c7bca00, uint64_t* BUF_IDX_61641c7bca00, HASHTABLE_INSERT HT_61641c7bca00, DBI32Type* customer__c_custkey, DBStringType* customer__c_mktsegment, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_mktsegment = customer__c_mktsegment[tid];
if (!(evaluatePredicate(reg_customer__c_mktsegment, "BUILDING", Predicate::eq))) return;
uint64_t KEY_61641c7bca00 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_61641c7bca00 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_61641c7bca00 = atomicAdd((int*)BUF_IDX_61641c7bca00, 1);
HT_61641c7bca00.insert(cuco::pair{KEY_61641c7bca00, buf_idx_61641c7bca00});
BUF_61641c7bca00[buf_idx_61641c7bca00 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_61641c7a9f70(uint64_t* BUF_61641c7bca00, uint64_t* COUNT61641c7bc640, HASHTABLE_PROBE HT_61641c7bca00, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 9204, Predicate::lt))) return;
uint64_t KEY_61641c7bca00 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_61641c7bca00 |= reg_orders__o_custkey;
//Probe Hash table
HT_61641c7bca00.for_each(KEY_61641c7bca00, [&] __device__ (auto const SLOT_61641c7bca00) {

auto const [slot_first61641c7bca00, slot_second61641c7bca00] = SLOT_61641c7bca00;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT61641c7bc640, 1);
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_61641c7a9f70(uint64_t* BUF_61641c7bc640, uint64_t* BUF_61641c7bca00, uint64_t* BUF_IDX_61641c7bc640, HASHTABLE_INSERT HT_61641c7bc640, HASHTABLE_PROBE HT_61641c7bca00, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 9204, Predicate::lt))) return;
uint64_t KEY_61641c7bca00 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_61641c7bca00 |= reg_orders__o_custkey;
//Probe Hash table
HT_61641c7bca00.for_each(KEY_61641c7bca00, [&] __device__ (auto const SLOT_61641c7bca00) {
auto const [slot_first61641c7bca00, slot_second61641c7bca00] = SLOT_61641c7bca00;
if (!(true)) return;
uint64_t KEY_61641c7bc640 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_61641c7bc640 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_61641c7bc640 = atomicAdd((int*)BUF_IDX_61641c7bc640, 1);
HT_61641c7bc640.insert(cuco::pair{KEY_61641c7bc640, buf_idx_61641c7bc640});
BUF_61641c7bc640[buf_idx_61641c7bc640 * 2 + 0] = BUF_61641c7bca00[slot_second61641c7bca00 * 1 + 0];
BUF_61641c7bc640[buf_idx_61641c7bc640 * 2 + 1] = tid;
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_61641c7c9300(uint64_t* BUF_61641c7bc640, HASHTABLE_INSERT HT_61641c778dc0, HASHTABLE_PROBE HT_61641c7bc640, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9204, Predicate::gt))) return;
uint64_t KEY_61641c7bc640 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_61641c7bc640 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_61641c7bc640.for_each(KEY_61641c7bc640, [&] __device__ (auto const SLOT_61641c7bc640) {

auto const [slot_first61641c7bc640, slot_second61641c7bc640] = SLOT_61641c7bc640;
if (!(true)) return;
uint64_t KEY_61641c778dc0 = 0;

KEY_61641c778dc0 |= reg_lineitem__l_orderkey;
//Create aggregation hash table
HT_61641c778dc0.insert(cuco::pair{KEY_61641c778dc0, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_61641c7c9300(uint64_t* BUF_61641c7bc640, HASHTABLE_FIND HT_61641c778dc0, HASHTABLE_PROBE HT_61641c7bc640, DBI32Type* KEY_61641c778dc0lineitem__l_orderkey, DBDecimalType* aggr0__tmp_attr0, DBDateType* aggr__o_orderdate, DBI32Type* aggr__o_shippriority, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, size_t lineitem_size, DBDateType* orders__o_orderdate, DBI32Type* orders__o_shippriority) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9204, Predicate::gt))) return;
uint64_t KEY_61641c7bc640 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_61641c7bc640 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_61641c7bc640.for_each(KEY_61641c7bc640, [&] __device__ (auto const SLOT_61641c7bc640) {
auto const [slot_first61641c7bc640, slot_second61641c7bc640] = SLOT_61641c7bc640;
if (!(true)) return;
uint64_t KEY_61641c778dc0 = 0;

KEY_61641c778dc0 |= reg_lineitem__l_orderkey;
//Aggregate in hashtable
auto buf_idx_61641c778dc0 = HT_61641c778dc0.find(KEY_61641c778dc0)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_61641c778dc0], reg_map0__tmp_attr1);
auto reg_orders__o_shippriority = orders__o_shippriority[BUF_61641c7bc640[slot_second61641c7bc640 * 2 + 1]];
aggregate_any(&aggr__o_shippriority[buf_idx_61641c778dc0], reg_orders__o_shippriority);
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_61641c7bc640[slot_second61641c7bc640 * 2 + 1]];
aggregate_any(&aggr__o_orderdate[buf_idx_61641c778dc0], reg_orders__o_orderdate);
KEY_61641c778dc0lineitem__l_orderkey[buf_idx_61641c778dc0] = reg_lineitem__l_orderkey;
});
}
__global__ void count_61641c7d56e0(uint64_t* COUNT61641c7585e0, size_t COUNT61641c778dc0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT61641c778dc0) return;
//Materialize count
atomicAdd((int*)COUNT61641c7585e0, 1);
}
__global__ void main_61641c7d56e0(size_t COUNT61641c778dc0, DBDecimalType* MAT61641c7585e0aggr0__tmp_attr0, DBDateType* MAT61641c7585e0aggr__o_orderdate, DBI32Type* MAT61641c7585e0aggr__o_shippriority, DBI32Type* MAT61641c7585e0lineitem__l_orderkey, uint64_t* MAT_IDX61641c7585e0, DBDecimalType* aggr0__tmp_attr0, DBDateType* aggr__o_orderdate, DBI32Type* aggr__o_shippriority, DBI32Type* lineitem__l_orderkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT61641c778dc0) return;
//Materialize buffers
auto mat_idx61641c7585e0 = atomicAdd((int*)MAT_IDX61641c7585e0, 1);
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];
MAT61641c7585e0lineitem__l_orderkey[mat_idx61641c7585e0] = reg_lineitem__l_orderkey;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT61641c7585e0aggr0__tmp_attr0[mat_idx61641c7585e0] = reg_aggr0__tmp_attr0;
auto reg_aggr__o_orderdate = aggr__o_orderdate[tid];
MAT61641c7585e0aggr__o_orderdate[mat_idx61641c7585e0] = reg_aggr__o_orderdate;
auto reg_aggr__o_shippriority = aggr__o_shippriority[tid];
MAT61641c7585e0aggr__o_shippriority[mat_idx61641c7585e0] = reg_aggr__o_shippriority;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map) {
//Materialize count
uint64_t* d_COUNT61641c7bca00;
hipMalloc(&d_COUNT61641c7bca00, sizeof(uint64_t));
hipMemset(d_COUNT61641c7bca00, 0, sizeof(uint64_t));
count_61641c7a9ca0<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT61641c7bca00, d_customer__c_mktsegment, customer_size);
uint64_t COUNT61641c7bca00;
hipMemcpy(&COUNT61641c7bca00, d_COUNT61641c7bca00, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_61641c7bca00;
hipMalloc(&d_BUF_IDX_61641c7bca00, sizeof(uint64_t));
hipMemset(d_BUF_IDX_61641c7bca00, 0, sizeof(uint64_t));
uint64_t* d_BUF_61641c7bca00;
hipMalloc(&d_BUF_61641c7bca00, sizeof(uint64_t) * COUNT61641c7bca00 * 1);
auto d_HT_61641c7bca00 = cuco::experimental::static_multimap{ (int)COUNT61641c7bca00*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_61641c7a9ca0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_61641c7bca00, d_BUF_IDX_61641c7bca00, d_HT_61641c7bca00.ref(cuco::insert), d_customer__c_custkey, d_customer__c_mktsegment, customer_size);
//Materialize count
uint64_t* d_COUNT61641c7bc640;
hipMalloc(&d_COUNT61641c7bc640, sizeof(uint64_t));
hipMemset(d_COUNT61641c7bc640, 0, sizeof(uint64_t));
count_61641c7a9f70<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_61641c7bca00, d_COUNT61641c7bc640, d_HT_61641c7bca00.ref(cuco::for_each), d_orders__o_custkey, d_orders__o_orderdate, orders_size);
uint64_t COUNT61641c7bc640;
hipMemcpy(&COUNT61641c7bc640, d_COUNT61641c7bc640, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_61641c7bc640;
hipMalloc(&d_BUF_IDX_61641c7bc640, sizeof(uint64_t));
hipMemset(d_BUF_IDX_61641c7bc640, 0, sizeof(uint64_t));
uint64_t* d_BUF_61641c7bc640;
hipMalloc(&d_BUF_61641c7bc640, sizeof(uint64_t) * COUNT61641c7bc640 * 2);
auto d_HT_61641c7bc640 = cuco::experimental::static_multimap{ (int)COUNT61641c7bc640*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_61641c7a9f70<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_61641c7bc640, d_BUF_61641c7bca00, d_BUF_IDX_61641c7bc640, d_HT_61641c7bc640.ref(cuco::insert), d_HT_61641c7bca00.ref(cuco::for_each), d_orders__o_custkey, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
//Create aggregation hash table
auto d_HT_61641c778dc0 = cuco::static_map{ (int)355555*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_61641c7c9300<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_61641c7bc640, d_HT_61641c778dc0.ref(cuco::insert), d_HT_61641c7bc640.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_shipdate, lineitem_size);
size_t COUNT61641c778dc0 = d_HT_61641c778dc0.size();
thrust::device_vector<int64_t> keys_61641c778dc0(COUNT61641c778dc0), vals_61641c778dc0(COUNT61641c778dc0);
d_HT_61641c778dc0.retrieve_all(keys_61641c778dc0.begin(), vals_61641c778dc0.begin());
d_HT_61641c778dc0.clear();
int64_t* raw_keys61641c778dc0 = thrust::raw_pointer_cast(keys_61641c778dc0.data());
insertKeys<<<std::ceil((float)COUNT61641c778dc0/32.), 32>>>(raw_keys61641c778dc0, d_HT_61641c778dc0.ref(cuco::insert), COUNT61641c778dc0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT61641c778dc0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT61641c778dc0);
DBI32Type* d_aggr__o_shippriority;
hipMalloc(&d_aggr__o_shippriority, sizeof(DBI32Type) * COUNT61641c778dc0);
hipMemset(d_aggr__o_shippriority, 0, sizeof(DBI32Type) * COUNT61641c778dc0);
DBDateType* d_aggr__o_orderdate;
hipMalloc(&d_aggr__o_orderdate, sizeof(DBDateType) * COUNT61641c778dc0);
hipMemset(d_aggr__o_orderdate, 0, sizeof(DBDateType) * COUNT61641c778dc0);
DBI32Type* d_KEY_61641c778dc0lineitem__l_orderkey;
hipMalloc(&d_KEY_61641c778dc0lineitem__l_orderkey, sizeof(DBI32Type) * COUNT61641c778dc0);
hipMemset(d_KEY_61641c778dc0lineitem__l_orderkey, 0, sizeof(DBI32Type) * COUNT61641c778dc0);
main_61641c7c9300<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_61641c7bc640, d_HT_61641c778dc0.ref(cuco::find), d_HT_61641c7bc640.ref(cuco::for_each), d_KEY_61641c778dc0lineitem__l_orderkey, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, lineitem_size, d_orders__o_orderdate, d_orders__o_shippriority);
//Materialize count
uint64_t* d_COUNT61641c7585e0;
hipMalloc(&d_COUNT61641c7585e0, sizeof(uint64_t));
hipMemset(d_COUNT61641c7585e0, 0, sizeof(uint64_t));
count_61641c7d56e0<<<std::ceil((float)COUNT61641c778dc0/32.), 32>>>(d_COUNT61641c7585e0, COUNT61641c778dc0);
uint64_t COUNT61641c7585e0;
hipMemcpy(&COUNT61641c7585e0, d_COUNT61641c7585e0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX61641c7585e0;
hipMalloc(&d_MAT_IDX61641c7585e0, sizeof(uint64_t));
hipMemset(d_MAT_IDX61641c7585e0, 0, sizeof(uint64_t));
auto MAT61641c7585e0lineitem__l_orderkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT61641c7585e0);
DBI32Type* d_MAT61641c7585e0lineitem__l_orderkey;
hipMalloc(&d_MAT61641c7585e0lineitem__l_orderkey, sizeof(DBI32Type) * COUNT61641c7585e0);
auto MAT61641c7585e0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT61641c7585e0);
DBDecimalType* d_MAT61641c7585e0aggr0__tmp_attr0;
hipMalloc(&d_MAT61641c7585e0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT61641c7585e0);
auto MAT61641c7585e0aggr__o_orderdate = (DBDateType*)malloc(sizeof(DBDateType) * COUNT61641c7585e0);
DBDateType* d_MAT61641c7585e0aggr__o_orderdate;
hipMalloc(&d_MAT61641c7585e0aggr__o_orderdate, sizeof(DBDateType) * COUNT61641c7585e0);
auto MAT61641c7585e0aggr__o_shippriority = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT61641c7585e0);
DBI32Type* d_MAT61641c7585e0aggr__o_shippriority;
hipMalloc(&d_MAT61641c7585e0aggr__o_shippriority, sizeof(DBI32Type) * COUNT61641c7585e0);
main_61641c7d56e0<<<std::ceil((float)COUNT61641c778dc0/32.), 32>>>(COUNT61641c778dc0, d_MAT61641c7585e0aggr0__tmp_attr0, d_MAT61641c7585e0aggr__o_orderdate, d_MAT61641c7585e0aggr__o_shippriority, d_MAT61641c7585e0lineitem__l_orderkey, d_MAT_IDX61641c7585e0, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_KEY_61641c778dc0lineitem__l_orderkey);
hipMemcpy(MAT61641c7585e0lineitem__l_orderkey, d_MAT61641c7585e0lineitem__l_orderkey, sizeof(DBI32Type) * COUNT61641c7585e0, hipMemcpyDeviceToHost);
hipMemcpy(MAT61641c7585e0aggr0__tmp_attr0, d_MAT61641c7585e0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT61641c7585e0, hipMemcpyDeviceToHost);
hipMemcpy(MAT61641c7585e0aggr__o_orderdate, d_MAT61641c7585e0aggr__o_orderdate, sizeof(DBDateType) * COUNT61641c7585e0, hipMemcpyDeviceToHost);
hipMemcpy(MAT61641c7585e0aggr__o_shippriority, d_MAT61641c7585e0aggr__o_shippriority, sizeof(DBI32Type) * COUNT61641c7585e0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT61641c7585e0; i++) { std::cout << MAT61641c7585e0lineitem__l_orderkey[i] << "\t";
std::cout << MAT61641c7585e0aggr0__tmp_attr0[i] << "\t";
std::cout << MAT61641c7585e0aggr__o_orderdate[i] << "\t";
std::cout << MAT61641c7585e0aggr__o_shippriority[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_61641c7bca00);
hipFree(d_BUF_IDX_61641c7bca00);
hipFree(d_COUNT61641c7bca00);
hipFree(d_BUF_61641c7bc640);
hipFree(d_BUF_IDX_61641c7bc640);
hipFree(d_COUNT61641c7bc640);
hipFree(d_KEY_61641c778dc0lineitem__l_orderkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr__o_orderdate);
hipFree(d_aggr__o_shippriority);
hipFree(d_COUNT61641c7585e0);
hipFree(d_MAT61641c7585e0aggr0__tmp_attr0);
hipFree(d_MAT61641c7585e0aggr__o_orderdate);
hipFree(d_MAT61641c7585e0aggr__o_shippriority);
hipFree(d_MAT61641c7585e0lineitem__l_orderkey);
hipFree(d_MAT_IDX61641c7585e0);
free(MAT61641c7585e0aggr0__tmp_attr0);
free(MAT61641c7585e0aggr__o_orderdate);
free(MAT61641c7585e0aggr__o_shippriority);
free(MAT61641c7585e0lineitem__l_orderkey);
}