#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5acbf86a3a40(uint64_t* COUNT5acbf86bec60, DBStringType* customer__c_mktsegment, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_mktsegment = customer__c_mktsegment[tid];
if (!(evaluatePredicate(reg_customer__c_mktsegment, "BUILDING", Predicate::eq))) return;
//Materialize count
atomicAdd((int*)COUNT5acbf86bec60, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5acbf86a3a40(uint64_t* BUF_5acbf86bec60, uint64_t* BUF_IDX_5acbf86bec60, HASHTABLE_INSERT HT_5acbf86bec60, DBI32Type* customer__c_custkey, DBStringType* customer__c_mktsegment, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_mktsegment = customer__c_mktsegment[tid];
if (!(evaluatePredicate(reg_customer__c_mktsegment, "BUILDING", Predicate::eq))) return;
uint64_t KEY_5acbf86bec60 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5acbf86bec60 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_5acbf86bec60 = atomicAdd((int*)BUF_IDX_5acbf86bec60, 1);
HT_5acbf86bec60.insert(cuco::pair{KEY_5acbf86bec60, buf_idx_5acbf86bec60});
BUF_5acbf86bec60[buf_idx_5acbf86bec60 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5acbf86a3d10(uint64_t* BUF_5acbf86bec60, uint64_t* COUNT5acbf86b6480, HASHTABLE_PROBE HT_5acbf86bec60, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 9204, Predicate::lt))) return;
uint64_t KEY_5acbf86bec60 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_5acbf86bec60 |= reg_orders__o_custkey;
//Probe Hash table
HT_5acbf86bec60.for_each(KEY_5acbf86bec60, [&] __device__ (auto const SLOT_5acbf86bec60) {

auto const [slot_first5acbf86bec60, slot_second5acbf86bec60] = SLOT_5acbf86bec60;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5acbf86b6480, 1);
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_5acbf86a3d10(uint64_t* BUF_5acbf86b6480, uint64_t* BUF_5acbf86bec60, uint64_t* BUF_IDX_5acbf86b6480, HASHTABLE_INSERT HT_5acbf86b6480, HASHTABLE_PROBE HT_5acbf86bec60, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 9204, Predicate::lt))) return;
uint64_t KEY_5acbf86bec60 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_5acbf86bec60 |= reg_orders__o_custkey;
//Probe Hash table
HT_5acbf86bec60.for_each(KEY_5acbf86bec60, [&] __device__ (auto const SLOT_5acbf86bec60) {
auto const [slot_first5acbf86bec60, slot_second5acbf86bec60] = SLOT_5acbf86bec60;
if (!(true)) return;
uint64_t KEY_5acbf86b6480 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_5acbf86b6480 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_5acbf86b6480 = atomicAdd((int*)BUF_IDX_5acbf86b6480, 1);
HT_5acbf86b6480.insert(cuco::pair{KEY_5acbf86b6480, buf_idx_5acbf86b6480});
BUF_5acbf86b6480[buf_idx_5acbf86b6480 * 2 + 0] = BUF_5acbf86bec60[slot_second5acbf86bec60 * 1 + 0];
BUF_5acbf86b6480[buf_idx_5acbf86b6480 * 2 + 1] = tid;
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5acbf86c3270(uint64_t* BUF_5acbf86b6480, HASHTABLE_INSERT HT_5acbf8671e00, HASHTABLE_PROBE HT_5acbf86b6480, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9204, Predicate::gt))) return;
uint64_t KEY_5acbf86b6480 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5acbf86b6480 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5acbf86b6480.for_each(KEY_5acbf86b6480, [&] __device__ (auto const SLOT_5acbf86b6480) {

auto const [slot_first5acbf86b6480, slot_second5acbf86b6480] = SLOT_5acbf86b6480;
if (!(true)) return;
uint64_t KEY_5acbf8671e00 = 0;

KEY_5acbf8671e00 |= reg_lineitem__l_orderkey;
//Create aggregation hash table
HT_5acbf8671e00.insert(cuco::pair{KEY_5acbf8671e00, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5acbf86c3270(uint64_t* BUF_5acbf86b6480, HASHTABLE_FIND HT_5acbf8671e00, HASHTABLE_PROBE HT_5acbf86b6480, DBI32Type* KEY_5acbf8671e00lineitem__l_orderkey, DBDecimalType* aggr0__tmp_attr0, DBDateType* aggr__o_orderdate, DBI32Type* aggr__o_shippriority, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, size_t lineitem_size, DBDateType* orders__o_orderdate, DBI32Type* orders__o_shippriority) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9204, Predicate::gt))) return;
uint64_t KEY_5acbf86b6480 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5acbf86b6480 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5acbf86b6480.for_each(KEY_5acbf86b6480, [&] __device__ (auto const SLOT_5acbf86b6480) {
auto const [slot_first5acbf86b6480, slot_second5acbf86b6480] = SLOT_5acbf86b6480;
if (!(true)) return;
uint64_t KEY_5acbf8671e00 = 0;

KEY_5acbf8671e00 |= reg_lineitem__l_orderkey;
//Aggregate in hashtable
auto buf_idx_5acbf8671e00 = HT_5acbf8671e00.find(KEY_5acbf8671e00)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5acbf8671e00], reg_map0__tmp_attr1);
auto reg_orders__o_shippriority = orders__o_shippriority[BUF_5acbf86b6480[slot_second5acbf86b6480 * 2 + 1]];
aggregate_any(&aggr__o_shippriority[buf_idx_5acbf8671e00], reg_orders__o_shippriority);
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_5acbf86b6480[slot_second5acbf86b6480 * 2 + 1]];
aggregate_any(&aggr__o_orderdate[buf_idx_5acbf8671e00], reg_orders__o_orderdate);
KEY_5acbf8671e00lineitem__l_orderkey[buf_idx_5acbf8671e00] = reg_lineitem__l_orderkey;
});
}
__global__ void count_5acbf86cddb0(uint64_t* COUNT5acbf8658a70, size_t COUNT5acbf8671e00) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5acbf8671e00) return;
//Materialize count
atomicAdd((int*)COUNT5acbf8658a70, 1);
}
__global__ void main_5acbf86cddb0(size_t COUNT5acbf8671e00, DBDecimalType* MAT5acbf8658a70aggr0__tmp_attr0, DBDateType* MAT5acbf8658a70aggr__o_orderdate, DBI32Type* MAT5acbf8658a70aggr__o_shippriority, DBI32Type* MAT5acbf8658a70lineitem__l_orderkey, uint64_t* MAT_IDX5acbf8658a70, DBDecimalType* aggr0__tmp_attr0, DBDateType* aggr__o_orderdate, DBI32Type* aggr__o_shippriority, DBI32Type* lineitem__l_orderkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5acbf8671e00) return;
//Materialize buffers
auto mat_idx5acbf8658a70 = atomicAdd((int*)MAT_IDX5acbf8658a70, 1);
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];
MAT5acbf8658a70lineitem__l_orderkey[mat_idx5acbf8658a70] = reg_lineitem__l_orderkey;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5acbf8658a70aggr0__tmp_attr0[mat_idx5acbf8658a70] = reg_aggr0__tmp_attr0;
auto reg_aggr__o_orderdate = aggr__o_orderdate[tid];
MAT5acbf8658a70aggr__o_orderdate[mat_idx5acbf8658a70] = reg_aggr__o_orderdate;
auto reg_aggr__o_shippriority = aggr__o_shippriority[tid];
MAT5acbf8658a70aggr__o_shippriority[mat_idx5acbf8658a70] = reg_aggr__o_shippriority;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT5acbf86bec60;
hipMalloc(&d_COUNT5acbf86bec60, sizeof(uint64_t));
hipMemset(d_COUNT5acbf86bec60, 0, sizeof(uint64_t));
count_5acbf86a3a40<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT5acbf86bec60, d_customer__c_mktsegment, customer_size);
uint64_t COUNT5acbf86bec60;
hipMemcpy(&COUNT5acbf86bec60, d_COUNT5acbf86bec60, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5acbf86bec60;
hipMalloc(&d_BUF_IDX_5acbf86bec60, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5acbf86bec60, 0, sizeof(uint64_t));
uint64_t* d_BUF_5acbf86bec60;
hipMalloc(&d_BUF_5acbf86bec60, sizeof(uint64_t) * COUNT5acbf86bec60 * 1);
auto d_HT_5acbf86bec60 = cuco::experimental::static_multimap{ (int)COUNT5acbf86bec60*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5acbf86a3a40<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5acbf86bec60, d_BUF_IDX_5acbf86bec60, d_HT_5acbf86bec60.ref(cuco::insert), d_customer__c_custkey, d_customer__c_mktsegment, customer_size);
//Materialize count
uint64_t* d_COUNT5acbf86b6480;
hipMalloc(&d_COUNT5acbf86b6480, sizeof(uint64_t));
hipMemset(d_COUNT5acbf86b6480, 0, sizeof(uint64_t));
count_5acbf86a3d10<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5acbf86bec60, d_COUNT5acbf86b6480, d_HT_5acbf86bec60.ref(cuco::for_each), d_orders__o_custkey, d_orders__o_orderdate, orders_size);
uint64_t COUNT5acbf86b6480;
hipMemcpy(&COUNT5acbf86b6480, d_COUNT5acbf86b6480, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5acbf86b6480;
hipMalloc(&d_BUF_IDX_5acbf86b6480, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5acbf86b6480, 0, sizeof(uint64_t));
uint64_t* d_BUF_5acbf86b6480;
hipMalloc(&d_BUF_5acbf86b6480, sizeof(uint64_t) * COUNT5acbf86b6480 * 2);
auto d_HT_5acbf86b6480 = cuco::experimental::static_multimap{ (int)COUNT5acbf86b6480*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5acbf86a3d10<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5acbf86b6480, d_BUF_5acbf86bec60, d_BUF_IDX_5acbf86b6480, d_HT_5acbf86b6480.ref(cuco::insert), d_HT_5acbf86bec60.ref(cuco::for_each), d_orders__o_custkey, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
//Create aggregation hash table
auto d_HT_5acbf8671e00 = cuco::static_map{ (int)355555*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5acbf86c3270<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5acbf86b6480, d_HT_5acbf8671e00.ref(cuco::insert), d_HT_5acbf86b6480.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_shipdate, lineitem_size);
size_t COUNT5acbf8671e00 = d_HT_5acbf8671e00.size();
thrust::device_vector<int64_t> keys_5acbf8671e00(COUNT5acbf8671e00), vals_5acbf8671e00(COUNT5acbf8671e00);
d_HT_5acbf8671e00.retrieve_all(keys_5acbf8671e00.begin(), vals_5acbf8671e00.begin());
d_HT_5acbf8671e00.clear();
int64_t* raw_keys5acbf8671e00 = thrust::raw_pointer_cast(keys_5acbf8671e00.data());
insertKeys<<<std::ceil((float)COUNT5acbf8671e00/32.), 32>>>(raw_keys5acbf8671e00, d_HT_5acbf8671e00.ref(cuco::insert), COUNT5acbf8671e00);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5acbf8671e00);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5acbf8671e00);
DBI32Type* d_aggr__o_shippriority;
hipMalloc(&d_aggr__o_shippriority, sizeof(DBI32Type) * COUNT5acbf8671e00);
hipMemset(d_aggr__o_shippriority, 0, sizeof(DBI32Type) * COUNT5acbf8671e00);
DBDateType* d_aggr__o_orderdate;
hipMalloc(&d_aggr__o_orderdate, sizeof(DBDateType) * COUNT5acbf8671e00);
hipMemset(d_aggr__o_orderdate, 0, sizeof(DBDateType) * COUNT5acbf8671e00);
DBI32Type* d_KEY_5acbf8671e00lineitem__l_orderkey;
hipMalloc(&d_KEY_5acbf8671e00lineitem__l_orderkey, sizeof(DBI32Type) * COUNT5acbf8671e00);
hipMemset(d_KEY_5acbf8671e00lineitem__l_orderkey, 0, sizeof(DBI32Type) * COUNT5acbf8671e00);
main_5acbf86c3270<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5acbf86b6480, d_HT_5acbf8671e00.ref(cuco::find), d_HT_5acbf86b6480.ref(cuco::for_each), d_KEY_5acbf8671e00lineitem__l_orderkey, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, lineitem_size, d_orders__o_orderdate, d_orders__o_shippriority);
//Materialize count
uint64_t* d_COUNT5acbf8658a70;
hipMalloc(&d_COUNT5acbf8658a70, sizeof(uint64_t));
hipMemset(d_COUNT5acbf8658a70, 0, sizeof(uint64_t));
count_5acbf86cddb0<<<std::ceil((float)COUNT5acbf8671e00/32.), 32>>>(d_COUNT5acbf8658a70, COUNT5acbf8671e00);
uint64_t COUNT5acbf8658a70;
hipMemcpy(&COUNT5acbf8658a70, d_COUNT5acbf8658a70, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5acbf8658a70;
hipMalloc(&d_MAT_IDX5acbf8658a70, sizeof(uint64_t));
hipMemset(d_MAT_IDX5acbf8658a70, 0, sizeof(uint64_t));
auto MAT5acbf8658a70lineitem__l_orderkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5acbf8658a70);
DBI32Type* d_MAT5acbf8658a70lineitem__l_orderkey;
hipMalloc(&d_MAT5acbf8658a70lineitem__l_orderkey, sizeof(DBI32Type) * COUNT5acbf8658a70);
auto MAT5acbf8658a70aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5acbf8658a70);
DBDecimalType* d_MAT5acbf8658a70aggr0__tmp_attr0;
hipMalloc(&d_MAT5acbf8658a70aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5acbf8658a70);
auto MAT5acbf8658a70aggr__o_orderdate = (DBDateType*)malloc(sizeof(DBDateType) * COUNT5acbf8658a70);
DBDateType* d_MAT5acbf8658a70aggr__o_orderdate;
hipMalloc(&d_MAT5acbf8658a70aggr__o_orderdate, sizeof(DBDateType) * COUNT5acbf8658a70);
auto MAT5acbf8658a70aggr__o_shippriority = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5acbf8658a70);
DBI32Type* d_MAT5acbf8658a70aggr__o_shippriority;
hipMalloc(&d_MAT5acbf8658a70aggr__o_shippriority, sizeof(DBI32Type) * COUNT5acbf8658a70);
main_5acbf86cddb0<<<std::ceil((float)COUNT5acbf8671e00/32.), 32>>>(COUNT5acbf8671e00, d_MAT5acbf8658a70aggr0__tmp_attr0, d_MAT5acbf8658a70aggr__o_orderdate, d_MAT5acbf8658a70aggr__o_shippriority, d_MAT5acbf8658a70lineitem__l_orderkey, d_MAT_IDX5acbf8658a70, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_KEY_5acbf8671e00lineitem__l_orderkey);
hipMemcpy(MAT5acbf8658a70lineitem__l_orderkey, d_MAT5acbf8658a70lineitem__l_orderkey, sizeof(DBI32Type) * COUNT5acbf8658a70, hipMemcpyDeviceToHost);
hipMemcpy(MAT5acbf8658a70aggr0__tmp_attr0, d_MAT5acbf8658a70aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5acbf8658a70, hipMemcpyDeviceToHost);
hipMemcpy(MAT5acbf8658a70aggr__o_orderdate, d_MAT5acbf8658a70aggr__o_orderdate, sizeof(DBDateType) * COUNT5acbf8658a70, hipMemcpyDeviceToHost);
hipMemcpy(MAT5acbf8658a70aggr__o_shippriority, d_MAT5acbf8658a70aggr__o_shippriority, sizeof(DBI32Type) * COUNT5acbf8658a70, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5acbf8658a70; i++) { std::cout << MAT5acbf8658a70lineitem__l_orderkey[i] << "\t";
std::cout << MAT5acbf8658a70aggr0__tmp_attr0[i] << "\t";
std::cout << MAT5acbf8658a70aggr__o_orderdate[i] << "\t";
std::cout << MAT5acbf8658a70aggr__o_shippriority[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_5acbf86bec60);
hipFree(d_BUF_IDX_5acbf86bec60);
hipFree(d_COUNT5acbf86bec60);
hipFree(d_BUF_5acbf86b6480);
hipFree(d_BUF_IDX_5acbf86b6480);
hipFree(d_COUNT5acbf86b6480);
hipFree(d_KEY_5acbf8671e00lineitem__l_orderkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr__o_orderdate);
hipFree(d_aggr__o_shippriority);
hipFree(d_COUNT5acbf8658a70);
hipFree(d_MAT5acbf8658a70aggr0__tmp_attr0);
hipFree(d_MAT5acbf8658a70aggr__o_orderdate);
hipFree(d_MAT5acbf8658a70aggr__o_shippriority);
hipFree(d_MAT5acbf8658a70lineitem__l_orderkey);
hipFree(d_MAT_IDX5acbf8658a70);
free(MAT5acbf8658a70aggr0__tmp_attr0);
free(MAT5acbf8658a70aggr__o_orderdate);
free(MAT5acbf8658a70aggr__o_shippriority);
free(MAT5acbf8658a70lineitem__l_orderkey);
}