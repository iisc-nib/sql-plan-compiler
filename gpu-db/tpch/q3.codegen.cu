#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_1(uint64_t* COUNT0, DBStringType* customer__c_mktsegment, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_mktsegment = customer__c_mktsegment[tid];
if (!(evaluatePredicate(reg_customer__c_mktsegment, "BUILDING", Predicate::eq))) return;
//Materialize count
atomicAdd((int*)COUNT0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBI32Type* customer__c_custkey, DBStringType* customer__c_mktsegment, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_mktsegment = customer__c_mktsegment[tid];
if (!(evaluatePredicate(reg_customer__c_mktsegment, "BUILDING", Predicate::eq))) return;
uint64_t KEY_0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0, buf_idx_0});
BUF_0[buf_idx_0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_3(uint64_t* BUF_0, uint64_t* COUNT2, HASHTABLE_PROBE HT_0, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 9204, Predicate::lt))) return;
uint64_t KEY_0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_0 |= reg_orders__o_custkey;
//Probe Hash table
HT_0.for_each(KEY_0, [&] __device__ (auto const SLOT_0) {

auto const [slot_first0, slot_second0] = SLOT_0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT2, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_0, uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_PROBE HT_0, HASHTABLE_INSERT HT_2, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 9204, Predicate::lt))) return;
uint64_t KEY_0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_0 |= reg_orders__o_custkey;
//Probe Hash table
HT_0.for_each(KEY_0, [&] __device__ (auto const SLOT_0) {
auto const [slot_first0, slot_second0] = SLOT_0;
if (!(true)) return;
uint64_t KEY_2 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_2 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_2 = atomicAdd((int*)BUF_IDX_2, 1);
HT_2.insert(cuco::pair{KEY_2, buf_idx_2});
BUF_2[buf_idx_2 * 2 + 0] = BUF_0[slot_second0 * 1 + 0];
BUF_2[buf_idx_2 * 2 + 1] = tid;
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_5(uint64_t* BUF_2, HASHTABLE_PROBE HT_2, HASHTABLE_INSERT HT_4, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9204, Predicate::gt))) return;
uint64_t KEY_2 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_2 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_2.for_each(KEY_2, [&] __device__ (auto const SLOT_2) {

auto const [slot_first2, slot_second2] = SLOT_2;
if (!(true)) return;
uint64_t KEY_4 = 0;

KEY_4 |= reg_lineitem__l_orderkey;
//Create aggregation hash table
HT_4.insert(cuco::pair{KEY_4, 1});
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_5(uint64_t* BUF_2, HASHTABLE_PROBE HT_2, HASHTABLE_FIND HT_4, DBI32Type* KEY_4lineitem__l_orderkey, DBDecimalType* aggr0__tmp_attr0, DBDateType* aggr__o_orderdate, DBI32Type* aggr__o_shippriority, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, size_t lineitem_size, DBDateType* orders__o_orderdate, DBI32Type* orders__o_shippriority) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9204, Predicate::gt))) return;
uint64_t KEY_2 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_2 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_2.for_each(KEY_2, [&] __device__ (auto const SLOT_2) {
auto const [slot_first2, slot_second2] = SLOT_2;
if (!(true)) return;
uint64_t KEY_4 = 0;

KEY_4 |= reg_lineitem__l_orderkey;
//Aggregate in hashtable
auto buf_idx_4 = HT_4.find(KEY_4)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1.0) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_4], reg_map0__tmp_attr1);
auto reg_orders__o_shippriority = orders__o_shippriority[BUF_2[slot_second2 * 2 + 1]];
aggregate_any(&aggr__o_shippriority[buf_idx_4], reg_orders__o_shippriority);
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_2[slot_second2 * 2 + 1]];
aggregate_any(&aggr__o_orderdate[buf_idx_4], reg_orders__o_orderdate);
KEY_4lineitem__l_orderkey[buf_idx_4] = reg_lineitem__l_orderkey;
});
}
__global__ void count_7(size_t COUNT4, uint64_t* COUNT6) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT4) return;
//Materialize count
atomicAdd((int*)COUNT6, 1);
}
__global__ void main_7(size_t COUNT4, DBDecimalType* MAT6aggr0__tmp_attr0, DBDateType* MAT6aggr__o_orderdate, DBI32Type* MAT6aggr__o_shippriority, DBI32Type* MAT6lineitem__l_orderkey, uint64_t* MAT_IDX6, DBDecimalType* aggr0__tmp_attr0, DBDateType* aggr__o_orderdate, DBI32Type* aggr__o_shippriority, DBI32Type* lineitem__l_orderkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT4) return;
//Materialize buffers
auto mat_idx6 = atomicAdd((int*)MAT_IDX6, 1);
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];
MAT6lineitem__l_orderkey[mat_idx6] = reg_lineitem__l_orderkey;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT6aggr0__tmp_attr0[mat_idx6] = reg_aggr0__tmp_attr0;
auto reg_aggr__o_orderdate = aggr__o_orderdate[tid];
MAT6aggr__o_orderdate[mat_idx6] = reg_aggr__o_orderdate;
auto reg_aggr__o_shippriority = aggr__o_shippriority[tid];
MAT6aggr__o_shippriority[mat_idx6] = reg_aggr__o_shippriority;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map) {
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)customer_size/128.), 128>>>(d_COUNT0, d_customer__c_mktsegment, customer_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::experimental::static_multimap{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_customer__c_custkey, d_customer__c_mktsegment, customer_size);
//Materialize count
uint64_t* d_COUNT2;
hipMalloc(&d_COUNT2, sizeof(uint64_t));
hipMemset(d_COUNT2, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)orders_size/128.), 128>>>(d_BUF_0, d_COUNT2, d_HT_0.ref(cuco::for_each), d_orders__o_custkey, d_orders__o_orderdate, orders_size);
uint64_t COUNT2;
hipMemcpy(&COUNT2, d_COUNT2, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 2);
auto d_HT_2 = cuco::experimental::static_multimap{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)orders_size/128.), 128>>>(d_BUF_0, d_BUF_2, d_BUF_IDX_2, d_HT_0.ref(cuco::for_each), d_HT_2.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
//Create aggregation hash table
auto d_HT_4 = cuco::static_map{ (int)355555*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5<<<std::ceil((float)lineitem_size/128.), 128>>>(d_BUF_2, d_HT_2.ref(cuco::for_each), d_HT_4.ref(cuco::insert), d_lineitem__l_orderkey, d_lineitem__l_shipdate, lineitem_size);
size_t COUNT4 = d_HT_4.size();
thrust::device_vector<int64_t> keys_4(COUNT4), vals_4(COUNT4);
d_HT_4.retrieve_all(keys_4.begin(), vals_4.begin());
d_HT_4.clear();
int64_t* raw_keys4 = thrust::raw_pointer_cast(keys_4.data());
insertKeys<<<std::ceil((float)COUNT4/128.), 128>>>(raw_keys4, d_HT_4.ref(cuco::insert), COUNT4);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT4);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT4);
DBI32Type* d_aggr__o_shippriority;
hipMalloc(&d_aggr__o_shippriority, sizeof(DBI32Type) * COUNT4);
hipMemset(d_aggr__o_shippriority, 0, sizeof(DBI32Type) * COUNT4);
DBDateType* d_aggr__o_orderdate;
hipMalloc(&d_aggr__o_orderdate, sizeof(DBDateType) * COUNT4);
hipMemset(d_aggr__o_orderdate, 0, sizeof(DBDateType) * COUNT4);
DBI32Type* d_KEY_4lineitem__l_orderkey;
hipMalloc(&d_KEY_4lineitem__l_orderkey, sizeof(DBI32Type) * COUNT4);
hipMemset(d_KEY_4lineitem__l_orderkey, 0, sizeof(DBI32Type) * COUNT4);
main_5<<<std::ceil((float)lineitem_size/128.), 128>>>(d_BUF_2, d_HT_2.ref(cuco::for_each), d_HT_4.ref(cuco::find), d_KEY_4lineitem__l_orderkey, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, lineitem_size, d_orders__o_orderdate, d_orders__o_shippriority);
//Materialize count
uint64_t* d_COUNT6;
hipMalloc(&d_COUNT6, sizeof(uint64_t));
hipMemset(d_COUNT6, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)COUNT4/128.), 128>>>(COUNT4, d_COUNT6);
uint64_t COUNT6;
hipMemcpy(&COUNT6, d_COUNT6, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX6;
hipMalloc(&d_MAT_IDX6, sizeof(uint64_t));
hipMemset(d_MAT_IDX6, 0, sizeof(uint64_t));
auto MAT6lineitem__l_orderkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT6);
DBI32Type* d_MAT6lineitem__l_orderkey;
hipMalloc(&d_MAT6lineitem__l_orderkey, sizeof(DBI32Type) * COUNT6);
auto MAT6aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT6);
DBDecimalType* d_MAT6aggr0__tmp_attr0;
hipMalloc(&d_MAT6aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6);
auto MAT6aggr__o_orderdate = (DBDateType*)malloc(sizeof(DBDateType) * COUNT6);
DBDateType* d_MAT6aggr__o_orderdate;
hipMalloc(&d_MAT6aggr__o_orderdate, sizeof(DBDateType) * COUNT6);
auto MAT6aggr__o_shippriority = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT6);
DBI32Type* d_MAT6aggr__o_shippriority;
hipMalloc(&d_MAT6aggr__o_shippriority, sizeof(DBI32Type) * COUNT6);
main_7<<<std::ceil((float)COUNT4/128.), 128>>>(COUNT4, d_MAT6aggr0__tmp_attr0, d_MAT6aggr__o_orderdate, d_MAT6aggr__o_shippriority, d_MAT6lineitem__l_orderkey, d_MAT_IDX6, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_KEY_4lineitem__l_orderkey);
hipMemcpy(MAT6lineitem__l_orderkey, d_MAT6lineitem__l_orderkey, sizeof(DBI32Type) * COUNT6, hipMemcpyDeviceToHost);
hipMemcpy(MAT6aggr0__tmp_attr0, d_MAT6aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6, hipMemcpyDeviceToHost);
hipMemcpy(MAT6aggr__o_orderdate, d_MAT6aggr__o_orderdate, sizeof(DBDateType) * COUNT6, hipMemcpyDeviceToHost);
hipMemcpy(MAT6aggr__o_shippriority, d_MAT6aggr__o_shippriority, sizeof(DBI32Type) * COUNT6, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT6; i++) { std::cout << "" << MAT6lineitem__l_orderkey[i];
std::cout << "," << MAT6aggr0__tmp_attr0[i];
std::cout << "," << MAT6aggr__o_orderdate[i];
std::cout << "," << MAT6aggr__o_shippriority[i];
std::cout << std::endl; }
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_COUNT0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_COUNT2);
hipFree(d_KEY_4lineitem__l_orderkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr__o_orderdate);
hipFree(d_aggr__o_shippriority);
hipFree(d_COUNT6);
hipFree(d_MAT6aggr0__tmp_attr0);
hipFree(d_MAT6aggr__o_orderdate);
hipFree(d_MAT6aggr__o_shippriority);
hipFree(d_MAT6lineitem__l_orderkey);
hipFree(d_MAT_IDX6);
free(MAT6aggr0__tmp_attr0);
free(MAT6aggr__o_orderdate);
free(MAT6aggr__o_shippriority);
free(MAT6lineitem__l_orderkey);
}