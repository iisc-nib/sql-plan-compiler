#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5861716cf1b0(uint64_t* COUNT5861716e1bc0, DBStringType* customer__c_mktsegment, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_mktsegment = customer__c_mktsegment[tid];
if (!(evaluatePredicate(reg_customer__c_mktsegment, "BUILDING", Predicate::eq))) return;
//Materialize count
atomicAdd((int*)COUNT5861716e1bc0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5861716cf1b0(uint64_t* BUF_5861716e1bc0, uint64_t* BUF_IDX_5861716e1bc0, HASHTABLE_INSERT HT_5861716e1bc0, DBI32Type* customer__c_custkey, DBStringType* customer__c_mktsegment, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_mktsegment = customer__c_mktsegment[tid];
if (!(evaluatePredicate(reg_customer__c_mktsegment, "BUILDING", Predicate::eq))) return;
uint64_t KEY_5861716e1bc0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5861716e1bc0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_5861716e1bc0 = atomicAdd((int*)BUF_IDX_5861716e1bc0, 1);
HT_5861716e1bc0.insert(cuco::pair{KEY_5861716e1bc0, buf_idx_5861716e1bc0});
BUF_5861716e1bc0[buf_idx_5861716e1bc0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5861716cf480(uint64_t* BUF_5861716e1bc0, uint64_t* COUNT5861716e1f10, HASHTABLE_PROBE HT_5861716e1bc0, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 9204, Predicate::lt))) return;
uint64_t KEY_5861716e1bc0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_5861716e1bc0 |= reg_orders__o_custkey;
//Probe Hash table
HT_5861716e1bc0.for_each(KEY_5861716e1bc0, [&] __device__ (auto const SLOT_5861716e1bc0) {

auto const [slot_first5861716e1bc0, slot_second5861716e1bc0] = SLOT_5861716e1bc0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5861716e1f10, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5861716cf480(uint64_t* BUF_5861716e1bc0, uint64_t* BUF_5861716e1f10, uint64_t* BUF_IDX_5861716e1f10, HASHTABLE_PROBE HT_5861716e1bc0, HASHTABLE_INSERT HT_5861716e1f10, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 9204, Predicate::lt))) return;
uint64_t KEY_5861716e1bc0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_5861716e1bc0 |= reg_orders__o_custkey;
//Probe Hash table
HT_5861716e1bc0.for_each(KEY_5861716e1bc0, [&] __device__ (auto const SLOT_5861716e1bc0) {
auto const [slot_first5861716e1bc0, slot_second5861716e1bc0] = SLOT_5861716e1bc0;
if (!(true)) return;
uint64_t KEY_5861716e1f10 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_5861716e1f10 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_5861716e1f10 = atomicAdd((int*)BUF_IDX_5861716e1f10, 1);
HT_5861716e1f10.insert(cuco::pair{KEY_5861716e1f10, buf_idx_5861716e1f10});
BUF_5861716e1f10[buf_idx_5861716e1f10 * 2 + 0] = BUF_5861716e1bc0[slot_second5861716e1bc0 * 1 + 0];
BUF_5861716e1f10[buf_idx_5861716e1f10 * 2 + 1] = tid;
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5861716eef40(uint64_t* BUF_5861716e1f10, HASHTABLE_INSERT HT_58617169d2b0, HASHTABLE_PROBE HT_5861716e1f10, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9204, Predicate::gt))) return;
uint64_t KEY_5861716e1f10 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5861716e1f10 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5861716e1f10.for_each(KEY_5861716e1f10, [&] __device__ (auto const SLOT_5861716e1f10) {

auto const [slot_first5861716e1f10, slot_second5861716e1f10] = SLOT_5861716e1f10;
if (!(true)) return;
uint64_t KEY_58617169d2b0 = 0;

KEY_58617169d2b0 |= reg_lineitem__l_orderkey;
//Create aggregation hash table
HT_58617169d2b0.insert(cuco::pair{KEY_58617169d2b0, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5861716eef40(uint64_t* BUF_5861716e1f10, HASHTABLE_FIND HT_58617169d2b0, HASHTABLE_PROBE HT_5861716e1f10, DBI32Type* KEY_58617169d2b0lineitem__l_orderkey, DBDecimalType* aggr0__tmp_attr0, DBDateType* aggr__o_orderdate, DBI32Type* aggr__o_shippriority, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, size_t lineitem_size, DBDateType* orders__o_orderdate, DBI32Type* orders__o_shippriority) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9204, Predicate::gt))) return;
uint64_t KEY_5861716e1f10 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5861716e1f10 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5861716e1f10.for_each(KEY_5861716e1f10, [&] __device__ (auto const SLOT_5861716e1f10) {
auto const [slot_first5861716e1f10, slot_second5861716e1f10] = SLOT_5861716e1f10;
if (!(true)) return;
uint64_t KEY_58617169d2b0 = 0;

KEY_58617169d2b0 |= reg_lineitem__l_orderkey;
//Aggregate in hashtable
auto buf_idx_58617169d2b0 = HT_58617169d2b0.find(KEY_58617169d2b0)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_58617169d2b0], reg_map0__tmp_attr1);
auto reg_orders__o_shippriority = orders__o_shippriority[BUF_5861716e1f10[slot_second5861716e1f10 * 2 + 1]];
aggregate_any(&aggr__o_shippriority[buf_idx_58617169d2b0], reg_orders__o_shippriority);
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_5861716e1f10[slot_second5861716e1f10 * 2 + 1]];
aggregate_any(&aggr__o_orderdate[buf_idx_58617169d2b0], reg_orders__o_orderdate);
KEY_58617169d2b0lineitem__l_orderkey[buf_idx_58617169d2b0] = reg_lineitem__l_orderkey;
});
}
__global__ void count_5861716fb300(uint64_t* COUNT58617167be90, size_t COUNT58617169d2b0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT58617169d2b0) return;
//Materialize count
atomicAdd((int*)COUNT58617167be90, 1);
}
__global__ void main_5861716fb300(size_t COUNT58617169d2b0, DBDecimalType* MAT58617167be90aggr0__tmp_attr0, DBDateType* MAT58617167be90aggr__o_orderdate, DBI32Type* MAT58617167be90aggr__o_shippriority, DBI32Type* MAT58617167be90lineitem__l_orderkey, uint64_t* MAT_IDX58617167be90, DBDecimalType* aggr0__tmp_attr0, DBDateType* aggr__o_orderdate, DBI32Type* aggr__o_shippriority, DBI32Type* lineitem__l_orderkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT58617169d2b0) return;
//Materialize buffers
auto mat_idx58617167be90 = atomicAdd((int*)MAT_IDX58617167be90, 1);
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];
MAT58617167be90lineitem__l_orderkey[mat_idx58617167be90] = reg_lineitem__l_orderkey;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT58617167be90aggr0__tmp_attr0[mat_idx58617167be90] = reg_aggr0__tmp_attr0;
auto reg_aggr__o_orderdate = aggr__o_orderdate[tid];
MAT58617167be90aggr__o_orderdate[mat_idx58617167be90] = reg_aggr__o_orderdate;
auto reg_aggr__o_shippriority = aggr__o_shippriority[tid];
MAT58617167be90aggr__o_shippriority[mat_idx58617167be90] = reg_aggr__o_shippriority;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map) {
//Materialize count
uint64_t* d_COUNT5861716e1bc0;
hipMalloc(&d_COUNT5861716e1bc0, sizeof(uint64_t));
hipMemset(d_COUNT5861716e1bc0, 0, sizeof(uint64_t));
count_5861716cf1b0<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT5861716e1bc0, d_customer__c_mktsegment, customer_size);
uint64_t COUNT5861716e1bc0;
hipMemcpy(&COUNT5861716e1bc0, d_COUNT5861716e1bc0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5861716e1bc0;
hipMalloc(&d_BUF_IDX_5861716e1bc0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5861716e1bc0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5861716e1bc0;
hipMalloc(&d_BUF_5861716e1bc0, sizeof(uint64_t) * COUNT5861716e1bc0 * 1);
auto d_HT_5861716e1bc0 = cuco::experimental::static_multimap{ (int)COUNT5861716e1bc0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5861716cf1b0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5861716e1bc0, d_BUF_IDX_5861716e1bc0, d_HT_5861716e1bc0.ref(cuco::insert), d_customer__c_custkey, d_customer__c_mktsegment, customer_size);
//Materialize count
uint64_t* d_COUNT5861716e1f10;
hipMalloc(&d_COUNT5861716e1f10, sizeof(uint64_t));
hipMemset(d_COUNT5861716e1f10, 0, sizeof(uint64_t));
count_5861716cf480<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5861716e1bc0, d_COUNT5861716e1f10, d_HT_5861716e1bc0.ref(cuco::for_each), d_orders__o_custkey, d_orders__o_orderdate, orders_size);
uint64_t COUNT5861716e1f10;
hipMemcpy(&COUNT5861716e1f10, d_COUNT5861716e1f10, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5861716e1f10;
hipMalloc(&d_BUF_IDX_5861716e1f10, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5861716e1f10, 0, sizeof(uint64_t));
uint64_t* d_BUF_5861716e1f10;
hipMalloc(&d_BUF_5861716e1f10, sizeof(uint64_t) * COUNT5861716e1f10 * 2);
auto d_HT_5861716e1f10 = cuco::experimental::static_multimap{ (int)COUNT5861716e1f10*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5861716cf480<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5861716e1bc0, d_BUF_5861716e1f10, d_BUF_IDX_5861716e1f10, d_HT_5861716e1bc0.ref(cuco::for_each), d_HT_5861716e1f10.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
//Create aggregation hash table
auto d_HT_58617169d2b0 = cuco::static_map{ (int)355555*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5861716eef40<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5861716e1f10, d_HT_58617169d2b0.ref(cuco::insert), d_HT_5861716e1f10.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_shipdate, lineitem_size);
size_t COUNT58617169d2b0 = d_HT_58617169d2b0.size();
thrust::device_vector<int64_t> keys_58617169d2b0(COUNT58617169d2b0), vals_58617169d2b0(COUNT58617169d2b0);
d_HT_58617169d2b0.retrieve_all(keys_58617169d2b0.begin(), vals_58617169d2b0.begin());
d_HT_58617169d2b0.clear();
int64_t* raw_keys58617169d2b0 = thrust::raw_pointer_cast(keys_58617169d2b0.data());
insertKeys<<<std::ceil((float)COUNT58617169d2b0/32.), 32>>>(raw_keys58617169d2b0, d_HT_58617169d2b0.ref(cuco::insert), COUNT58617169d2b0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT58617169d2b0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT58617169d2b0);
DBI32Type* d_aggr__o_shippriority;
hipMalloc(&d_aggr__o_shippriority, sizeof(DBI32Type) * COUNT58617169d2b0);
hipMemset(d_aggr__o_shippriority, 0, sizeof(DBI32Type) * COUNT58617169d2b0);
DBDateType* d_aggr__o_orderdate;
hipMalloc(&d_aggr__o_orderdate, sizeof(DBDateType) * COUNT58617169d2b0);
hipMemset(d_aggr__o_orderdate, 0, sizeof(DBDateType) * COUNT58617169d2b0);
DBI32Type* d_KEY_58617169d2b0lineitem__l_orderkey;
hipMalloc(&d_KEY_58617169d2b0lineitem__l_orderkey, sizeof(DBI32Type) * COUNT58617169d2b0);
hipMemset(d_KEY_58617169d2b0lineitem__l_orderkey, 0, sizeof(DBI32Type) * COUNT58617169d2b0);
main_5861716eef40<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5861716e1f10, d_HT_58617169d2b0.ref(cuco::find), d_HT_5861716e1f10.ref(cuco::for_each), d_KEY_58617169d2b0lineitem__l_orderkey, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, lineitem_size, d_orders__o_orderdate, d_orders__o_shippriority);
//Materialize count
uint64_t* d_COUNT58617167be90;
hipMalloc(&d_COUNT58617167be90, sizeof(uint64_t));
hipMemset(d_COUNT58617167be90, 0, sizeof(uint64_t));
count_5861716fb300<<<std::ceil((float)COUNT58617169d2b0/32.), 32>>>(d_COUNT58617167be90, COUNT58617169d2b0);
uint64_t COUNT58617167be90;
hipMemcpy(&COUNT58617167be90, d_COUNT58617167be90, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX58617167be90;
hipMalloc(&d_MAT_IDX58617167be90, sizeof(uint64_t));
hipMemset(d_MAT_IDX58617167be90, 0, sizeof(uint64_t));
auto MAT58617167be90lineitem__l_orderkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT58617167be90);
DBI32Type* d_MAT58617167be90lineitem__l_orderkey;
hipMalloc(&d_MAT58617167be90lineitem__l_orderkey, sizeof(DBI32Type) * COUNT58617167be90);
auto MAT58617167be90aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT58617167be90);
DBDecimalType* d_MAT58617167be90aggr0__tmp_attr0;
hipMalloc(&d_MAT58617167be90aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT58617167be90);
auto MAT58617167be90aggr__o_orderdate = (DBDateType*)malloc(sizeof(DBDateType) * COUNT58617167be90);
DBDateType* d_MAT58617167be90aggr__o_orderdate;
hipMalloc(&d_MAT58617167be90aggr__o_orderdate, sizeof(DBDateType) * COUNT58617167be90);
auto MAT58617167be90aggr__o_shippriority = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT58617167be90);
DBI32Type* d_MAT58617167be90aggr__o_shippriority;
hipMalloc(&d_MAT58617167be90aggr__o_shippriority, sizeof(DBI32Type) * COUNT58617167be90);
main_5861716fb300<<<std::ceil((float)COUNT58617169d2b0/32.), 32>>>(COUNT58617169d2b0, d_MAT58617167be90aggr0__tmp_attr0, d_MAT58617167be90aggr__o_orderdate, d_MAT58617167be90aggr__o_shippriority, d_MAT58617167be90lineitem__l_orderkey, d_MAT_IDX58617167be90, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_KEY_58617169d2b0lineitem__l_orderkey);
hipMemcpy(MAT58617167be90lineitem__l_orderkey, d_MAT58617167be90lineitem__l_orderkey, sizeof(DBI32Type) * COUNT58617167be90, hipMemcpyDeviceToHost);
hipMemcpy(MAT58617167be90aggr0__tmp_attr0, d_MAT58617167be90aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT58617167be90, hipMemcpyDeviceToHost);
hipMemcpy(MAT58617167be90aggr__o_orderdate, d_MAT58617167be90aggr__o_orderdate, sizeof(DBDateType) * COUNT58617167be90, hipMemcpyDeviceToHost);
hipMemcpy(MAT58617167be90aggr__o_shippriority, d_MAT58617167be90aggr__o_shippriority, sizeof(DBI32Type) * COUNT58617167be90, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT58617167be90; i++) { std::cout << MAT58617167be90lineitem__l_orderkey[i] << "\t";
std::cout << MAT58617167be90aggr0__tmp_attr0[i] << "\t";
std::cout << MAT58617167be90aggr__o_orderdate[i] << "\t";
std::cout << MAT58617167be90aggr__o_shippriority[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_5861716e1bc0);
hipFree(d_BUF_IDX_5861716e1bc0);
hipFree(d_COUNT5861716e1bc0);
hipFree(d_BUF_5861716e1f10);
hipFree(d_BUF_IDX_5861716e1f10);
hipFree(d_COUNT5861716e1f10);
hipFree(d_KEY_58617169d2b0lineitem__l_orderkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr__o_orderdate);
hipFree(d_aggr__o_shippriority);
hipFree(d_COUNT58617167be90);
hipFree(d_MAT58617167be90aggr0__tmp_attr0);
hipFree(d_MAT58617167be90aggr__o_orderdate);
hipFree(d_MAT58617167be90aggr__o_shippriority);
hipFree(d_MAT58617167be90lineitem__l_orderkey);
hipFree(d_MAT_IDX58617167be90);
free(MAT58617167be90aggr0__tmp_attr0);
free(MAT58617167be90aggr__o_orderdate);
free(MAT58617167be90aggr__o_shippriority);
free(MAT58617167be90lineitem__l_orderkey);
}