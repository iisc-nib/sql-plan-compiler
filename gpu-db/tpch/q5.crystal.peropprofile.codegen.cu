#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_3(uint64_t* COUNT4, DBStringType* region__r_name, size_t region_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_region__r_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
reg_region__r_name[ITEM] = region__r_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_region__r_name[ITEM], "ASIA", Predicate::eq);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT4, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_INSERT HT_4, int64_t* cycles_per_warp_main_3_join_build_4, int64_t* cycles_per_warp_main_3_selection_2, DBStringType* region__r_name, DBI32Type* region__r_regionkey, size_t region_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_region__r_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
reg_region__r_name[ITEM] = region__r_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_region__r_name[ITEM], "ASIA", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_region__r_regionkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
reg_region__r_regionkey[ITEM] = region__r_regionkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_region__r_regionkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_4 = atomicAdd((int*)BUF_IDX_4, 1);
HT_4.insert(cuco::pair{KEY_4[ITEM], buf_idx_4});
BUF_4[(buf_idx_4) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_build_4[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE>
__global__ void count_5(uint64_t* BUF_4, uint64_t* COUNT6, HASHTABLE_PROBE HT_4, DBI32Type* nation__n_regionkey, size_t nation_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_nation__n_regionkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_nation__n_regionkey[ITEM] = nation__n_regionkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_nation__n_regionkey[ITEM];
}
int64_t slot_second4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0; continue;}
slot_second4[ITEM] = SLOT_4->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT6, 1);
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_4, uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_PROBE HT_4, HASHTABLE_INSERT HT_6, int64_t* cycles_per_warp_main_5_join_build_6, int64_t* cycles_per_warp_main_5_join_probe_4, DBI32Type* nation__n_nationkey, DBI32Type* nation__n_regionkey, size_t nation_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_nation__n_regionkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_nation__n_regionkey[ITEM] = nation__n_regionkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_nation__n_regionkey[ITEM];
}
int64_t slot_second4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0; continue;}
slot_second4[ITEM] = SLOT_4->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_join_probe_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_nation__n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_nation__n_nationkey[ITEM] = nation__n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_nation__n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_6 = atomicAdd((int*)BUF_IDX_6, 1);
HT_6.insert(cuco::pair{KEY_6[ITEM], buf_idx_6});
BUF_6[(buf_idx_6) * 2 + 0] = BUF_4[slot_second4[ITEM] * 1 + 0];
BUF_6[(buf_idx_6) * 2 + 1] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_join_build_6[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE>
__global__ void count_7(uint64_t* BUF_6, uint64_t* COUNT8, HASHTABLE_PROBE HT_6, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_customer__c_nationkey[ITEM];
}
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT8, 1);
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_7(uint64_t* BUF_6, uint64_t* BUF_8, uint64_t* BUF_IDX_8, HASHTABLE_PROBE HT_6, HASHTABLE_INSERT HT_8, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size, int64_t* cycles_per_warp_main_7_join_build_8, int64_t* cycles_per_warp_main_7_join_probe_6) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_customer__c_nationkey[ITEM];
}
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_probe_6[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_8 = atomicAdd((int*)BUF_IDX_8, 1);
HT_8.insert(cuco::pair{KEY_8[ITEM], buf_idx_8});
BUF_8[(buf_idx_8) * 3 + 0] = ITEM*TB + tid;
BUF_8[(buf_idx_8) * 3 + 1] = BUF_6[slot_second6[ITEM] * 2 + 0];
BUF_8[(buf_idx_8) * 3 + 2] = BUF_6[slot_second6[ITEM] * 2 + 1];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_build_8[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE>
__global__ void count_1(uint64_t* BUF_8, uint64_t* COUNT9, HASHTABLE_PROBE HT_8, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_orders__o_orderdate[ITEM], 8766, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate[ITEM], 9131, Predicate::lt);
}
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_custkey[ITEM] = orders__o_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_orders__o_custkey[ITEM];
}
int64_t slot_second8[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_8 = HT_8.find(KEY_8[ITEM]);
if (SLOT_8 == HT_8.end()) {selection_flags[ITEM] = 0; continue;}
slot_second8[ITEM] = SLOT_8->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT9, 1);
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_8, uint64_t* BUF_9, uint64_t* BUF_IDX_9, HASHTABLE_PROBE HT_8, HASHTABLE_INSERT HT_9, int64_t* cycles_per_warp_main_1_join_build_9, int64_t* cycles_per_warp_main_1_join_probe_8, int64_t* cycles_per_warp_main_1_selection_0, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_orders__o_orderdate[ITEM], 8766, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate[ITEM], 9131, Predicate::lt);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_custkey[ITEM] = orders__o_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_orders__o_custkey[ITEM];
}
int64_t slot_second8[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_8 = HT_8.find(KEY_8[ITEM]);
if (SLOT_8 == HT_8.end()) {selection_flags[ITEM] = 0; continue;}
slot_second8[ITEM] = SLOT_8->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_probe_8[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_9[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_9[ITEM] = 0;
KEY_9[ITEM] |= reg_orders__o_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_9 = atomicAdd((int*)BUF_IDX_9, 1);
HT_9.insert(cuco::pair{KEY_9[ITEM], buf_idx_9});
BUF_9[(buf_idx_9) * 4 + 0] = BUF_8[slot_second8[ITEM] * 3 + 0];
BUF_9[(buf_idx_9) * 4 + 1] = ITEM*TB + tid;
BUF_9[(buf_idx_9) * 4 + 2] = BUF_8[slot_second8[ITEM] * 3 + 1];
BUF_9[(buf_idx_9) * 4 + 3] = BUF_8[slot_second8[ITEM] * 3 + 2];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_build_9[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_12(uint64_t* COUNT11, size_t supplier_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT11, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_12(uint64_t* BUF_11, uint64_t* BUF_IDX_11, HASHTABLE_INSERT HT_11, int64_t* cycles_per_warp_main_12_join_build_11, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_11[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
DBI32Type reg_supplier__s_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_nationkey[ITEM] = supplier__s_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_11[ITEM] = 0;
KEY_11[ITEM] |= reg_supplier__s_suppkey[ITEM];
KEY_11[ITEM] <<= 32;
KEY_11[ITEM] |= reg_supplier__s_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_11 = atomicAdd((int*)BUF_IDX_11, 1);
HT_11.insert(cuco::pair{KEY_11[ITEM], buf_idx_11});
BUF_11[(buf_idx_11) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_12_join_build_11[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_10(uint64_t* BUF_11, uint64_t* BUF_9, HASHTABLE_PROBE HT_11, HASHTABLE_INSERT HT_14, HASHTABLE_PROBE HT_9, DBI32Type* customer__c_nationkey, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_9[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_9[ITEM] = 0;
KEY_9[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
int64_t slot_second9[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_9 = HT_9.find(KEY_9[ITEM]);
if (SLOT_9 == HT_9.end()) {selection_flags[ITEM] = 0; continue;}
slot_second9[ITEM] = SLOT_9->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_11[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_suppkey[ITEM] = lineitem__l_suppkey[ITEM*TB + tid];
}
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[BUF_9[slot_second9[ITEM] * 4 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_11[ITEM] = 0;
KEY_11[ITEM] |= reg_lineitem__l_suppkey[ITEM];
KEY_11[ITEM] <<= 32;
KEY_11[ITEM] |= reg_customer__c_nationkey[ITEM];
}
int64_t slot_second11[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_11 = HT_11.find(KEY_11[ITEM]);
if (SLOT_11 == HT_11.end()) {selection_flags[ITEM] = 0; continue;}
slot_second11[ITEM] = SLOT_11->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_14[ITEMS_PER_THREAD];
DBI16Type reg_nation__n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_nation__n_name_encoded[ITEM] = nation__n_name_encoded[BUF_9[slot_second9[ITEM] * 4 + 3]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_14[ITEM] = 0;
KEY_14[ITEM] |= reg_nation__n_name_encoded[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_14.insert(cuco::pair{KEY_14[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_10(uint64_t* BUF_11, uint64_t* BUF_9, HASHTABLE_PROBE HT_11, HASHTABLE_FIND HT_14, HASHTABLE_PROBE HT_9, DBI16Type* KEY_14nation__n_name_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* customer__c_nationkey, int64_t* cycles_per_warp_main_10_aggregation_14, int64_t* cycles_per_warp_main_10_join_probe_11, int64_t* cycles_per_warp_main_10_join_probe_9, int64_t* cycles_per_warp_main_10_map_13, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_9[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_9[ITEM] = 0;
KEY_9[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
int64_t slot_second9[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_9 = HT_9.find(KEY_9[ITEM]);
if (SLOT_9 == HT_9.end()) {selection_flags[ITEM] = 0; continue;}
slot_second9[ITEM] = SLOT_9->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_10_join_probe_9[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_11[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_suppkey[ITEM] = lineitem__l_suppkey[ITEM*TB + tid];
}
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[BUF_9[slot_second9[ITEM] * 4 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_11[ITEM] = 0;
KEY_11[ITEM] |= reg_lineitem__l_suppkey[ITEM];
KEY_11[ITEM] <<= 32;
KEY_11[ITEM] |= reg_customer__c_nationkey[ITEM];
}
int64_t slot_second11[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_11 = HT_11.find(KEY_11[ITEM]);
if (SLOT_11 == HT_11.end()) {selection_flags[ITEM] = 0; continue;}
slot_second11[ITEM] = SLOT_11->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_10_join_probe_11[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_10_map_13[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_14[ITEMS_PER_THREAD];
DBI16Type reg_nation__n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_nation__n_name_encoded[ITEM] = nation__n_name_encoded[BUF_9[slot_second9[ITEM] * 4 + 3]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_14[ITEM] = 0;
KEY_14[ITEM] |= reg_nation__n_name_encoded[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_14 = HT_14.find(KEY_14[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr0[buf_idx_14], reg_map0__tmp_attr1[ITEM]);
KEY_14nation__n_name_encoded[buf_idx_14] = reg_nation__n_name_encoded[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_10_aggregation_14[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_16(size_t COUNT14, uint64_t* COUNT15) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT14); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT15, 1);
}
}
__global__ void main_16(size_t COUNT14, DBDecimalType* MAT15aggr0__tmp_attr0, DBI16Type* MAT15nation__n_name_encoded, uint64_t* MAT_IDX15, DBDecimalType* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_16_materialize_15, DBI16Type* nation__n_name_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBI16Type reg_nation__n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT14); ++ITEM) {
reg_nation__n_name_encoded[ITEM] = nation__n_name_encoded[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT14); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT14); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx15 = atomicAdd((int*)MAT_IDX15, 1);
MAT15nation__n_name_encoded[mat_idx15] = reg_nation__n_name_encoded[ITEM];
MAT15aggr0__tmp_attr0[mat_idx15] = reg_aggr0__tmp_attr0[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_16_materialize_15[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_3_selection_2;
auto main_3_selection_2_cpw_size = std::ceil((float)region_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_selection_2, sizeof(int64_t) * main_3_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_3_selection_2, -1, sizeof(int64_t) * main_3_selection_2_cpw_size);
//Materialize count
uint64_t* d_COUNT4;
hipMalloc(&d_COUNT4, sizeof(uint64_t));
hipMemset(d_COUNT4, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)region_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT4, d_region__r_name, region_size);
uint64_t COUNT4;
hipMemcpy(&COUNT4, d_COUNT4, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_3_join_build_4;
auto main_3_join_build_4_cpw_size = std::ceil((float)region_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_build_4, sizeof(int64_t) * main_3_join_build_4_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_build_4, -1, sizeof(int64_t) * main_3_join_build_4_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 1);
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)region_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_BUF_IDX_4, d_HT_4.ref(cuco::insert), d_cycles_per_warp_main_3_join_build_4, d_cycles_per_warp_main_3_selection_2, d_region__r_name, d_region__r_regionkey, region_size);
int64_t* cycles_per_warp_main_3_selection_2 = (int64_t*)malloc(sizeof(int64_t) * main_3_selection_2_cpw_size);
hipMemcpy(cycles_per_warp_main_3_selection_2, d_cycles_per_warp_main_3_selection_2, sizeof(int64_t) * main_3_selection_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_selection_2 ";
for (auto i=0ull; i < main_3_selection_2_cpw_size; i++) std::cout << cycles_per_warp_main_3_selection_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_join_build_4 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_build_4_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_build_4, d_cycles_per_warp_main_3_join_build_4, sizeof(int64_t) * main_3_join_build_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_build_4 ";
for (auto i=0ull; i < main_3_join_build_4_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_build_4[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_5_join_probe_4;
auto main_5_join_probe_4_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_join_probe_4, sizeof(int64_t) * main_5_join_probe_4_cpw_size);
hipMemset(d_cycles_per_warp_main_5_join_probe_4, -1, sizeof(int64_t) * main_5_join_probe_4_cpw_size);
//Materialize count
uint64_t* d_COUNT6;
hipMalloc(&d_COUNT6, sizeof(uint64_t));
hipMemset(d_COUNT6, 0, sizeof(uint64_t));
count_5<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_COUNT6, d_HT_4.ref(cuco::find), d_nation__n_regionkey, nation_size);
uint64_t COUNT6;
hipMemcpy(&COUNT6, d_COUNT6, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_5_join_build_6;
auto main_5_join_build_6_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_join_build_6, sizeof(int64_t) * main_5_join_build_6_cpw_size);
hipMemset(d_cycles_per_warp_main_5_join_build_6, -1, sizeof(int64_t) * main_5_join_build_6_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 2);
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_BUF_6, d_BUF_IDX_6, d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::insert), d_cycles_per_warp_main_5_join_build_6, d_cycles_per_warp_main_5_join_probe_4, d_nation__n_nationkey, d_nation__n_regionkey, nation_size);
int64_t* cycles_per_warp_main_5_join_probe_4 = (int64_t*)malloc(sizeof(int64_t) * main_5_join_probe_4_cpw_size);
hipMemcpy(cycles_per_warp_main_5_join_probe_4, d_cycles_per_warp_main_5_join_probe_4, sizeof(int64_t) * main_5_join_probe_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_join_probe_4 ";
for (auto i=0ull; i < main_5_join_probe_4_cpw_size; i++) std::cout << cycles_per_warp_main_5_join_probe_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_join_build_6 = (int64_t*)malloc(sizeof(int64_t) * main_5_join_build_6_cpw_size);
hipMemcpy(cycles_per_warp_main_5_join_build_6, d_cycles_per_warp_main_5_join_build_6, sizeof(int64_t) * main_5_join_build_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_join_build_6 ";
for (auto i=0ull; i < main_5_join_build_6_cpw_size; i++) std::cout << cycles_per_warp_main_5_join_build_6[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_7_join_probe_6;
auto main_7_join_probe_6_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_probe_6, sizeof(int64_t) * main_7_join_probe_6_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_probe_6, -1, sizeof(int64_t) * main_7_join_probe_6_cpw_size);
//Materialize count
uint64_t* d_COUNT8;
hipMalloc(&d_COUNT8, sizeof(uint64_t));
hipMemset(d_COUNT8, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_6, d_COUNT8, d_HT_6.ref(cuco::find), d_customer__c_nationkey, customer_size);
uint64_t COUNT8;
hipMemcpy(&COUNT8, d_COUNT8, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_7_join_build_8;
auto main_7_join_build_8_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_build_8, sizeof(int64_t) * main_7_join_build_8_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_build_8, -1, sizeof(int64_t) * main_7_join_build_8_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_8;
hipMalloc(&d_BUF_IDX_8, sizeof(uint64_t));
hipMemset(d_BUF_IDX_8, 0, sizeof(uint64_t));
uint64_t* d_BUF_8;
hipMalloc(&d_BUF_8, sizeof(uint64_t) * COUNT8 * 3);
auto d_HT_8 = cuco::static_map{ (int)COUNT8*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_6, d_BUF_8, d_BUF_IDX_8, d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::insert), d_customer__c_custkey, d_customer__c_nationkey, customer_size, d_cycles_per_warp_main_7_join_build_8, d_cycles_per_warp_main_7_join_probe_6);
int64_t* cycles_per_warp_main_7_join_probe_6 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_probe_6_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_probe_6, d_cycles_per_warp_main_7_join_probe_6, sizeof(int64_t) * main_7_join_probe_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_probe_6 ";
for (auto i=0ull; i < main_7_join_probe_6_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_probe_6[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_join_build_8 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_build_8_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_build_8, d_cycles_per_warp_main_7_join_build_8, sizeof(int64_t) * main_7_join_build_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_build_8 ";
for (auto i=0ull; i < main_7_join_build_8_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_build_8[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_join_probe_8;
auto main_1_join_probe_8_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_probe_8, sizeof(int64_t) * main_1_join_probe_8_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_probe_8, -1, sizeof(int64_t) * main_1_join_probe_8_cpw_size);
//Materialize count
uint64_t* d_COUNT9;
hipMalloc(&d_COUNT9, sizeof(uint64_t));
hipMemset(d_COUNT9, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_8, d_COUNT9, d_HT_8.ref(cuco::find), d_orders__o_custkey, d_orders__o_orderdate, orders_size);
uint64_t COUNT9;
hipMemcpy(&COUNT9, d_COUNT9, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_1_join_build_9;
auto main_1_join_build_9_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_build_9, sizeof(int64_t) * main_1_join_build_9_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_build_9, -1, sizeof(int64_t) * main_1_join_build_9_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_9;
hipMalloc(&d_BUF_IDX_9, sizeof(uint64_t));
hipMemset(d_BUF_IDX_9, 0, sizeof(uint64_t));
uint64_t* d_BUF_9;
hipMalloc(&d_BUF_9, sizeof(uint64_t) * COUNT9 * 4);
auto d_HT_9 = cuco::static_map{ (int)COUNT9*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_8, d_BUF_9, d_BUF_IDX_9, d_HT_8.ref(cuco::find), d_HT_9.ref(cuco::insert), d_cycles_per_warp_main_1_join_build_9, d_cycles_per_warp_main_1_join_probe_8, d_cycles_per_warp_main_1_selection_0, d_orders__o_custkey, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_probe_8 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_probe_8_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_probe_8, d_cycles_per_warp_main_1_join_probe_8, sizeof(int64_t) * main_1_join_probe_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_probe_8 ";
for (auto i=0ull; i < main_1_join_probe_8_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_probe_8[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_build_9 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_build_9_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_build_9, d_cycles_per_warp_main_1_join_build_9, sizeof(int64_t) * main_1_join_build_9_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_build_9 ";
for (auto i=0ull; i < main_1_join_build_9_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_build_9[i] << " ";
std::cout << std::endl;
//Materialize count
uint64_t* d_COUNT11;
hipMalloc(&d_COUNT11, sizeof(uint64_t));
hipMemset(d_COUNT11, 0, sizeof(uint64_t));
count_12<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT11, supplier_size);
uint64_t COUNT11;
hipMemcpy(&COUNT11, d_COUNT11, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_12_join_build_11;
auto main_12_join_build_11_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_12_join_build_11, sizeof(int64_t) * main_12_join_build_11_cpw_size);
hipMemset(d_cycles_per_warp_main_12_join_build_11, -1, sizeof(int64_t) * main_12_join_build_11_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_11;
hipMalloc(&d_BUF_IDX_11, sizeof(uint64_t));
hipMemset(d_BUF_IDX_11, 0, sizeof(uint64_t));
uint64_t* d_BUF_11;
hipMalloc(&d_BUF_11, sizeof(uint64_t) * COUNT11 * 1);
auto d_HT_11 = cuco::static_map{ (int)COUNT11*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_12<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_11, d_BUF_IDX_11, d_HT_11.ref(cuco::insert), d_cycles_per_warp_main_12_join_build_11, d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
int64_t* cycles_per_warp_main_12_join_build_11 = (int64_t*)malloc(sizeof(int64_t) * main_12_join_build_11_cpw_size);
hipMemcpy(cycles_per_warp_main_12_join_build_11, d_cycles_per_warp_main_12_join_build_11, sizeof(int64_t) * main_12_join_build_11_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_12_join_build_11 ";
for (auto i=0ull; i < main_12_join_build_11_cpw_size; i++) std::cout << cycles_per_warp_main_12_join_build_11[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_10_join_probe_9;
auto main_10_join_probe_9_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_10_join_probe_9, sizeof(int64_t) * main_10_join_probe_9_cpw_size);
hipMemset(d_cycles_per_warp_main_10_join_probe_9, -1, sizeof(int64_t) * main_10_join_probe_9_cpw_size);
int64_t* d_cycles_per_warp_main_10_join_probe_11;
auto main_10_join_probe_11_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_10_join_probe_11, sizeof(int64_t) * main_10_join_probe_11_cpw_size);
hipMemset(d_cycles_per_warp_main_10_join_probe_11, -1, sizeof(int64_t) * main_10_join_probe_11_cpw_size);
int64_t* d_cycles_per_warp_main_10_map_13;
auto main_10_map_13_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_10_map_13, sizeof(int64_t) * main_10_map_13_cpw_size);
hipMemset(d_cycles_per_warp_main_10_map_13, -1, sizeof(int64_t) * main_10_map_13_cpw_size);
//Create aggregation hash table
auto d_HT_14 = cuco::static_map{ (int)22857*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_10<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_11, d_BUF_9, d_HT_11.ref(cuco::find), d_HT_14.ref(cuco::insert), d_HT_9.ref(cuco::find), d_customer__c_nationkey, d_lineitem__l_orderkey, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded);
size_t COUNT14 = d_HT_14.size();
thrust::device_vector<int64_t> keys_14(COUNT14), vals_14(COUNT14);
d_HT_14.retrieve_all(keys_14.begin(), vals_14.begin());
d_HT_14.clear();
int64_t* raw_keys14 = thrust::raw_pointer_cast(keys_14.data());
insertKeys<<<std::ceil((float)COUNT14/128.), 128>>>(raw_keys14, d_HT_14.ref(cuco::insert), COUNT14);
int64_t* d_cycles_per_warp_main_10_aggregation_14;
auto main_10_aggregation_14_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_10_aggregation_14, sizeof(int64_t) * main_10_aggregation_14_cpw_size);
hipMemset(d_cycles_per_warp_main_10_aggregation_14, -1, sizeof(int64_t) * main_10_aggregation_14_cpw_size);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT14);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT14);
DBI16Type* d_KEY_14nation__n_name_encoded;
hipMalloc(&d_KEY_14nation__n_name_encoded, sizeof(DBI16Type) * COUNT14);
hipMemset(d_KEY_14nation__n_name_encoded, 0, sizeof(DBI16Type) * COUNT14);
main_10<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_11, d_BUF_9, d_HT_11.ref(cuco::find), d_HT_14.ref(cuco::find), d_HT_9.ref(cuco::find), d_KEY_14nation__n_name_encoded, d_aggr0__tmp_attr0, d_customer__c_nationkey, d_cycles_per_warp_main_10_aggregation_14, d_cycles_per_warp_main_10_join_probe_11, d_cycles_per_warp_main_10_join_probe_9, d_cycles_per_warp_main_10_map_13, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded);
int64_t* cycles_per_warp_main_10_join_probe_9 = (int64_t*)malloc(sizeof(int64_t) * main_10_join_probe_9_cpw_size);
hipMemcpy(cycles_per_warp_main_10_join_probe_9, d_cycles_per_warp_main_10_join_probe_9, sizeof(int64_t) * main_10_join_probe_9_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_10_join_probe_9 ";
for (auto i=0ull; i < main_10_join_probe_9_cpw_size; i++) std::cout << cycles_per_warp_main_10_join_probe_9[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_10_join_probe_11 = (int64_t*)malloc(sizeof(int64_t) * main_10_join_probe_11_cpw_size);
hipMemcpy(cycles_per_warp_main_10_join_probe_11, d_cycles_per_warp_main_10_join_probe_11, sizeof(int64_t) * main_10_join_probe_11_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_10_join_probe_11 ";
for (auto i=0ull; i < main_10_join_probe_11_cpw_size; i++) std::cout << cycles_per_warp_main_10_join_probe_11[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_10_map_13 = (int64_t*)malloc(sizeof(int64_t) * main_10_map_13_cpw_size);
hipMemcpy(cycles_per_warp_main_10_map_13, d_cycles_per_warp_main_10_map_13, sizeof(int64_t) * main_10_map_13_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_10_map_13 ";
for (auto i=0ull; i < main_10_map_13_cpw_size; i++) std::cout << cycles_per_warp_main_10_map_13[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_10_aggregation_14 = (int64_t*)malloc(sizeof(int64_t) * main_10_aggregation_14_cpw_size);
hipMemcpy(cycles_per_warp_main_10_aggregation_14, d_cycles_per_warp_main_10_aggregation_14, sizeof(int64_t) * main_10_aggregation_14_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_10_aggregation_14 ";
for (auto i=0ull; i < main_10_aggregation_14_cpw_size; i++) std::cout << cycles_per_warp_main_10_aggregation_14[i] << " ";
std::cout << std::endl;
//Materialize count
uint64_t* d_COUNT15;
hipMalloc(&d_COUNT15, sizeof(uint64_t));
hipMemset(d_COUNT15, 0, sizeof(uint64_t));
count_16<<<std::ceil((float)COUNT14/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT14, d_COUNT15);
uint64_t COUNT15;
hipMemcpy(&COUNT15, d_COUNT15, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_16_materialize_15;
auto main_16_materialize_15_cpw_size = std::ceil((float)COUNT14/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_16_materialize_15, sizeof(int64_t) * main_16_materialize_15_cpw_size);
hipMemset(d_cycles_per_warp_main_16_materialize_15, -1, sizeof(int64_t) * main_16_materialize_15_cpw_size);
//Materialize buffers
uint64_t* d_MAT_IDX15;
hipMalloc(&d_MAT_IDX15, sizeof(uint64_t));
hipMemset(d_MAT_IDX15, 0, sizeof(uint64_t));
auto MAT15nation__n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT15);
DBI16Type* d_MAT15nation__n_name_encoded;
hipMalloc(&d_MAT15nation__n_name_encoded, sizeof(DBI16Type) * COUNT15);
auto MAT15aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT15);
DBDecimalType* d_MAT15aggr0__tmp_attr0;
hipMalloc(&d_MAT15aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT15);
main_16<<<std::ceil((float)COUNT14/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT14, d_MAT15aggr0__tmp_attr0, d_MAT15nation__n_name_encoded, d_MAT_IDX15, d_aggr0__tmp_attr0, d_cycles_per_warp_main_16_materialize_15, d_KEY_14nation__n_name_encoded);
hipMemcpy(MAT15nation__n_name_encoded, d_MAT15nation__n_name_encoded, sizeof(DBI16Type) * COUNT15, hipMemcpyDeviceToHost);
hipMemcpy(MAT15aggr0__tmp_attr0, d_MAT15aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT15, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_16_materialize_15 = (int64_t*)malloc(sizeof(int64_t) * main_16_materialize_15_cpw_size);
hipMemcpy(cycles_per_warp_main_16_materialize_15, d_cycles_per_warp_main_16_materialize_15, sizeof(int64_t) * main_16_materialize_15_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_16_materialize_15 ";
for (auto i=0ull; i < main_16_materialize_15_cpw_size; i++) std::cout << cycles_per_warp_main_16_materialize_15[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
      size_t aux_mem = usedGpuMem() - used_mem;
      std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_COUNT4);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_COUNT6);
hipFree(d_BUF_8);
hipFree(d_BUF_IDX_8);
hipFree(d_COUNT8);
hipFree(d_BUF_9);
hipFree(d_BUF_IDX_9);
hipFree(d_COUNT9);
hipFree(d_BUF_11);
hipFree(d_BUF_IDX_11);
hipFree(d_COUNT11);
hipFree(d_KEY_14nation__n_name_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT15);
hipFree(d_MAT15aggr0__tmp_attr0);
hipFree(d_MAT15nation__n_name_encoded);
hipFree(d_MAT_IDX15);
free(MAT15aggr0__tmp_attr0);
free(MAT15nation__n_name_encoded);
}