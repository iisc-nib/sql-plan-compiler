#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <dirent.h>
#include <dlfcn.h>
#include <iomanip>
#include <chrono>
#include "dbruntime.h"

__global__ void sample(DBStringType *s1, DBStringType *s2, int *res)
{
    if (evaluatePredicate(s1[0], s2[0], Predicate::eq))
        atomicAdd(res, 1);
}

void runQuery(std::string &libPath)
{
    void *lib = dlopen(libPath.c_str(), RTLD_NOW);
    if (!lib)
    {
        fprintf(stderr, "%s\n", dlerror());
    }
    else
    {
        std::clog << "Opening the shared lib " << libPath << " was successful!" << std::endl;

        auto control = reinterpret_cast<void (*)(
            DBI32Type *,
            DBStringType *,
            DBI32Type *,
            DBStringType *,
            size_t,
            DBI32Type *,
            DBI32Type *,
            DBStringType *,
            DBStringType *,
            DBStringType *,
            DBDecimalType *,
            DBStringType *,
            size_t,
            DBI32Type *,
            DBI32Type *,
            DBI32Type *,
            DBDecimalType *,
            DBStringType *,
            size_t,
            DBI32Type *,
            DBStringType *,
            DBStringType *,
            DBStringType *,
            DBStringType *,
            DBI32Type *,
            DBStringType *,
            DBDecimalType *,
            DBStringType *,
            size_t,
            DBI32Type *,
            DBI32Type *,
            DBI32Type *,
            DBI64Type *,
            DBDecimalType *,
            DBDecimalType *,
            DBDecimalType *,
            DBDecimalType *,
            DBCharType *,
            DBCharType *,
            DBI32Type *,
            DBI32Type *,
            DBI32Type *,
            DBStringType *,
            DBStringType *,
            DBStringType *,
            size_t,
            DBI32Type *,
            DBCharType *,
            DBI32Type *,
            DBDecimalType *,
            DBI32Type *,
            DBStringType *,
            DBStringType *,
            DBI32Type *,
            DBStringType *,
            size_t,
            DBI32Type *,
            DBStringType *,
            DBStringType *,
            DBI32Type *,
            DBStringType *,
            DBDecimalType *,
            DBStringType *,
            DBStringType *,
            size_t,
            DBI32Type *,
            DBStringType *,
            DBStringType *,
            size_t,
            DBI16Type *,
            std::unordered_map<DBI16Type, std::string> &,
            std::unordered_map<DBI16Type, std::string> &,
            std::unordered_map<DBI16Type, std::string> &,
            DBI16Type*,
            std::unordered_map<DBI16Type, std::string> &,
            DBI16Type*,
            std::unordered_map<DBI16Type, std::string> &,
            DBI16Type*,
            std::unordered_map<DBI16Type, std::string> &,
            DBI16Type*,
            std::unordered_map<DBI16Type, std::string> &,
            DBI16Type*,
            std::unordered_map<DBI16Type, std::string> &,
            DBI16Type*,
            std::unordered_map<DBI16Type, std::string> &,
            DBI16Type*,
            std::unordered_map<DBI16Type, std::string> &,
            DBI16Type*,
            std::unordered_map<DBI16Type, std::string> &
        )>(dlsym(lib, "control"));
        auto start = std::chrono::high_resolution_clock::now();
        control(
            d_nation__n_nationkey,
            d_nation__n_name,
            d_nation__n_regionkey,
            d_nation__n_comment,
            nation_size,
            d_supplier__s_suppkey,
            d_supplier__s_nationkey,
            d_supplier__s_name,
            d_supplier__s_address,
            d_supplier__s_phone,
            d_supplier__s_acctbal,
            d_supplier__s_comment,
            supplier_size,
            d_partsupp__ps_suppkey,
            d_partsupp__ps_partkey,
            d_partsupp__ps_availqty,
            d_partsupp__ps_supplycost,
            d_partsupp__ps_comment,
            partsupp_size,
            d_part__p_partkey,
            d_part__p_name,
            d_part__p_mfgr,
            d_part__p_brand,
            d_part__p_type,
            d_part__p_size,
            d_part__p_container,
            d_part__p_retailprice,
            d_part__p_comment,
            part_size,
            d_lineitem__l_orderkey,
            d_lineitem__l_partkey,
            d_lineitem__l_suppkey,
            d_lineitem__l_linenumber,
            d_lineitem__l_quantity,
            d_lineitem__l_extendedprice,
            d_lineitem__l_discount,
            d_lineitem__l_tax,
            d_lineitem__l_returnflag,
            d_lineitem__l_linestatus,
            d_lineitem__l_shipdate,
            d_lineitem__l_commitdate,
            d_lineitem__l_receiptdate,
            d_lineitem__l_shipinstruct,
            d_lineitem__l_shipmode,
            d_lineitem__comments,
            lineitem_size,
            d_orders__o_orderkey,
            d_orders__o_orderstatus,
            d_orders__o_custkey,
            d_orders__o_totalprice,
            d_orders__o_orderdate,
            d_orders__o_orderpriority,
            d_orders__o_clerk,
            d_orders__o_shippriority,
            d_orders__o_comment,
            orders_size,
            d_customer__c_custkey,
            d_customer__c_name,
            d_customer__c_address,
            d_customer__c_nationkey,
            d_customer__c_phone,
            d_customer__c_acctbal,
            d_customer__c_mktsegment,
            d_customer__c_comment,
            customer_size,
            d_region__r_regionkey,
            d_region__r_name,
            d_region__r_comment,
            region_size,
            d_nation__n_name_encoded,
            nation__n_name_map,
            nation__n_name_map,
            nation__n_name_map,
            d_orders__o_orderpriority_encoded,
            orders__o_orderpriority_map,
            d_customer__c_name_encoded,
            customer__c_name_map,
            d_customer__c_comment_encoded,
            customer__c_comment_map,
            d_customer__c_phone_encoded,
            customer__c_phone_map,
            d_customer__c_address_encoded,
            customer__c_address_map,
            d_supplier__s_name_encoded,
            supplier__s_name_map,
            d_part__p_brand_encoded,
            part__p_brand_map,
            d_part__p_type_encoded,
            part__p_type_map
            );
        auto stop = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
        std::clog << "Query execution time: " << duration.count() / 1000. << "milliseconds.\n";
        dlclose(lib);
    }
}

std::string getQueryNumber(int argc, const char **argv)
{
    std::string queryNum;
    for (int32_t i = 0; i < argc; i++)
    {
        if (std::string(argv[i]) == "--query_num")
        {
            if (i + 1 < argc)
                queryNum = std::string(argv[i + 1]);
            break;
        }
    }
    return queryNum;
}

int main(int argc, const char **argv)
{
    std::string dbDir = getDataDir(argv, argc);
    auto queryNum = getQueryNumber(argc, argv);
    // std::cout << "Arg len: " << argc << std::endl;
    // assert(argc >= 4 && "Need 3 arguments, --data-dir <path to dir> <shared object query>");
    // const char *soFile = argv[3];
    initTpchDb(dbDir);

    std::cout << std::setprecision(12);

    // TODO(avinash): the runtime should listen for sql queries,
    //  compile it with the lingodb toolchain
    //  look for output.cu
    //  compile it with nvcc
    //  link it using dlopen, and execute the control function.

    // void *lib = dlopen("/media/ajayakar/space/src/sql-plan-compiler/gpu-db/tpch_auto_generated/"+ soFile, RTLD_LAZY);
    if (queryNum != "")
    {
        std::string libPath = "build/q" + queryNum + ".codegen.so";
        runQuery(libPath);
    }
    else
    {
        while (true)
        {
            std::string libPath;
            std::cout << "> ";
            std::cin >> libPath;
            if (libPath == "exit" || libPath == "quit" || libPath == "q")
                break;
            libPath = "build/q" + libPath + ".codegen.so"; // input as integer now for convinience
            runQuery(libPath);
        }
    }
}
