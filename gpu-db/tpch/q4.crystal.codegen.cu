#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_1(uint64_t* COUNT0, DBDateType* lineitem__l_commitdate, DBDateType* lineitem__l_receiptdate, size_t lineitem_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_lineitem__l_commitdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_commitdate[ITEM] = lineitem__l_commitdate[ITEM*TB + tid];
}
DBDateType reg_lineitem__l_receiptdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_receiptdate[ITEM] = lineitem__l_receiptdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_commitdate[ITEM], reg_lineitem__l_receiptdate[ITEM], Predicate::lt);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT0, 1);
}
}
template<typename HASHTABLE_INSERT_SJ>
__global__ void main_1(HASHTABLE_INSERT_SJ HT_0, DBDateType* lineitem__l_commitdate, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_receiptdate, size_t lineitem_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_lineitem__l_commitdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_commitdate[ITEM] = lineitem__l_commitdate[ITEM*TB + tid];
}
DBDateType reg_lineitem__l_receiptdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_receiptdate[ITEM] = lineitem__l_receiptdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_commitdate[ITEM], reg_lineitem__l_receiptdate[ITEM], Predicate::lt);
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_0.insert(cuco::pair{KEY_0[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE_SJ, typename HASHTABLE_INSERT>
__global__ void count_3(HASHTABLE_PROBE_SJ HT_0, HASHTABLE_INSERT HT_2, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, DBI16Type* orders__o_orderpriority_encoded, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_orders__o_orderdate[ITEM], 8582, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate[ITEM], 8674, Predicate::lt);
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_orders__o_orderkey[ITEM];
}
//Probe Hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0;}
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI16Type reg_orders__o_orderpriority_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderpriority_encoded[ITEM] = orders__o_orderpriority_encoded[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_orders__o_orderpriority_encoded[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_2.insert(cuco::pair{KEY_2[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE_SJ, typename HASHTABLE_FIND>
__global__ void main_3(HASHTABLE_PROBE_SJ HT_0, HASHTABLE_FIND HT_2, DBI16Type* KEY_2orders__o_orderpriority_encoded, DBI64Type* aggr0__tmp_attr0, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, DBI16Type* orders__o_orderpriority_encoded, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_orders__o_orderdate[ITEM], 8582, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate[ITEM], 8674, Predicate::lt);
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_orders__o_orderkey[ITEM];
}
//Probe Hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0;}
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI16Type reg_orders__o_orderpriority_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderpriority_encoded[ITEM] = orders__o_orderpriority_encoded[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_orders__o_orderpriority_encoded[ITEM];
}
//Aggregate in hashtable
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_2 = HT_2.find(KEY_2[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr0[buf_idx_2], 1);
KEY_2orders__o_orderpriority_encoded[buf_idx_2] = reg_orders__o_orderpriority_encoded[ITEM];
}
}
__global__ void count_5(size_t COUNT2, uint64_t* COUNT4) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT4, 1);
}
}
__global__ void main_5(size_t COUNT2, DBI64Type* MAT4aggr0__tmp_attr0, DBI16Type* MAT4orders__o_orderpriority_encoded, uint64_t* MAT_IDX4, DBI64Type* aggr0__tmp_attr0, DBI16Type* orders__o_orderpriority_encoded) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize buffers
DBI16Type reg_orders__o_orderpriority_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
reg_orders__o_orderpriority_encoded[ITEM] = orders__o_orderpriority_encoded[ITEM*TB + tid];
}
DBI64Type reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx4 = atomicAdd((int*)MAT_IDX4, 1);
MAT4orders__o_orderpriority_encoded[mat_idx4] = reg_orders__o_orderpriority_encoded[ITEM];
MAT4aggr0__tmp_attr0[mat_idx4] = reg_aggr0__tmp_attr0[ITEM];
}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto start = std::chrono::high_resolution_clock::now();
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT0, d_lineitem__l_commitdate, d_lineitem__l_receiptdate, lineitem_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_HT_0.ref(cuco::insert), d_lineitem__l_commitdate, d_lineitem__l_orderkey, d_lineitem__l_receiptdate, lineitem_size);
//Create aggregation hash table
auto d_HT_2 = cuco::static_map{ (int)51270*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_3<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::insert), d_orders__o_orderdate, d_orders__o_orderkey, d_orders__o_orderpriority_encoded, orders_size);
size_t COUNT2 = d_HT_2.size();
thrust::device_vector<int64_t> keys_2(COUNT2), vals_2(COUNT2);
d_HT_2.retrieve_all(keys_2.begin(), vals_2.begin());
d_HT_2.clear();
int64_t* raw_keys2 = thrust::raw_pointer_cast(keys_2.data());
insertKeys<<<std::ceil((float)COUNT2/128.), 128>>>(raw_keys2, d_HT_2.ref(cuco::insert), COUNT2);
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT2);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT2);
DBI16Type* d_KEY_2orders__o_orderpriority_encoded;
hipMalloc(&d_KEY_2orders__o_orderpriority_encoded, sizeof(DBI16Type) * COUNT2);
hipMemset(d_KEY_2orders__o_orderpriority_encoded, 0, sizeof(DBI16Type) * COUNT2);
main_3<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::find), d_KEY_2orders__o_orderpriority_encoded, d_aggr0__tmp_attr0, d_orders__o_orderdate, d_orders__o_orderkey, d_orders__o_orderpriority_encoded, orders_size);
//Materialize count
uint64_t* d_COUNT4;
hipMalloc(&d_COUNT4, sizeof(uint64_t));
hipMemset(d_COUNT4, 0, sizeof(uint64_t));
count_5<<<std::ceil((float)COUNT2/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT2, d_COUNT4);
uint64_t COUNT4;
hipMemcpy(&COUNT4, d_COUNT4, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX4;
hipMalloc(&d_MAT_IDX4, sizeof(uint64_t));
hipMemset(d_MAT_IDX4, 0, sizeof(uint64_t));
auto MAT4orders__o_orderpriority_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT4);
DBI16Type* d_MAT4orders__o_orderpriority_encoded;
hipMalloc(&d_MAT4orders__o_orderpriority_encoded, sizeof(DBI16Type) * COUNT4);
auto MAT4aggr0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT4);
DBI64Type* d_MAT4aggr0__tmp_attr0;
hipMalloc(&d_MAT4aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT4);
main_5<<<std::ceil((float)COUNT2/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT2, d_MAT4aggr0__tmp_attr0, d_MAT4orders__o_orderpriority_encoded, d_MAT_IDX4, d_aggr0__tmp_attr0, d_KEY_2orders__o_orderpriority_encoded);
hipMemcpy(MAT4orders__o_orderpriority_encoded, d_MAT4orders__o_orderpriority_encoded, sizeof(DBI16Type) * COUNT4, hipMemcpyDeviceToHost);
hipMemcpy(MAT4aggr0__tmp_attr0, d_MAT4aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT4, hipMemcpyDeviceToHost);
auto end = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < COUNT4; i++) { std::cout << "" << orders__o_orderpriority_map[MAT4orders__o_orderpriority_encoded[i]];
std::cout << "|" << MAT4aggr0__tmp_attr0[i];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
      size_t aux_mem = usedGpuMem() - used_mem;
      std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_COUNT0);
hipFree(d_KEY_2orders__o_orderpriority_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT4);
hipFree(d_MAT4aggr0__tmp_attr0);
hipFree(d_MAT4orders__o_orderpriority_encoded);
hipFree(d_MAT_IDX4);
free(MAT4aggr0__tmp_attr0);
free(MAT4orders__o_orderpriority_encoded);
}