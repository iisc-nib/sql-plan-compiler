#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_INSERT>
__global__ void count_1(HASHTABLE_INSERT HT_0, DBCharType* lineitem__l_linestatus, DBCharType* lineitem__l_returnflag, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_lineitem__l_shipdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipdate[ITEM] = lineitem__l_shipdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 10471, Predicate::lte);
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBCharType reg_lineitem__l_returnflag[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_returnflag[ITEM] = lineitem__l_returnflag[ITEM*TB + tid];
}
DBCharType reg_lineitem__l_linestatus[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_linestatus[ITEM] = lineitem__l_linestatus[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_lineitem__l_returnflag[ITEM];
KEY_0[ITEM] <<= 8;
KEY_0[ITEM] |= reg_lineitem__l_linestatus[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_0.insert(cuco::pair{KEY_0[ITEM], 1});
}
}
template<typename HASHTABLE_FIND>
__global__ void main_1(HASHTABLE_FIND HT_0, DBCharType* KEY_0lineitem__l_linestatus, DBCharType* KEY_0lineitem__l_returnflag, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr0__tmp_attr1, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, DBI64Type* aggr0__tmp_attr9, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, DBDecimalType* aggr_rw__rw2, DBI64Type* aggr_rw__rw3, DBDecimalType* aggr_rw__rw4, DBI64Type* aggr_rw__rw5, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBCharType* lineitem__l_linestatus, DBDecimalType* lineitem__l_quantity, DBCharType* lineitem__l_returnflag, DBDateType* lineitem__l_shipdate, DBDecimalType* lineitem__l_tax, size_t lineitem_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_lineitem__l_shipdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipdate[ITEM] = lineitem__l_shipdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 10471, Predicate::lte);
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBCharType reg_lineitem__l_returnflag[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_returnflag[ITEM] = lineitem__l_returnflag[ITEM*TB + tid];
}
DBCharType reg_lineitem__l_linestatus[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_linestatus[ITEM] = lineitem__l_linestatus[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_lineitem__l_returnflag[ITEM];
KEY_0[ITEM] <<= 8;
KEY_0[ITEM] |= reg_lineitem__l_linestatus[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_tax[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_tax[ITEM] = lineitem__l_tax[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr5[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr5[ITEM] = ((reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]))) * ((1.0) + (reg_lineitem__l_tax[ITEM]));
}
DBDecimalType reg_map0__tmp_attr3[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr3[ITEM] = (reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]));
}
DBDecimalType reg_lineitem__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_quantity[ITEM] = lineitem__l_quantity[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_0 = HT_0.find(KEY_0[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr9[buf_idx_0], 1);
aggregate_sum(&aggr0__tmp_attr4[buf_idx_0], reg_map0__tmp_attr5[ITEM]);
aggregate_sum(&aggr0__tmp_attr2[buf_idx_0], reg_map0__tmp_attr3[ITEM]);
aggregate_sum(&aggr0__tmp_attr1[buf_idx_0], reg_lineitem__l_extendedprice[ITEM]);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_0], reg_lineitem__l_quantity[ITEM]);
aggregate_sum(&aggr_rw__rw0[buf_idx_0], reg_lineitem__l_discount[ITEM]);
aggregate_sum(&aggr_rw__rw1[buf_idx_0], 1);
aggregate_sum(&aggr_rw__rw2[buf_idx_0], reg_lineitem__l_extendedprice[ITEM]);
aggregate_sum(&aggr_rw__rw3[buf_idx_0], 1);
aggregate_sum(&aggr_rw__rw4[buf_idx_0], reg_lineitem__l_quantity[ITEM]);
aggregate_sum(&aggr_rw__rw5[buf_idx_0], 1);
KEY_0lineitem__l_returnflag[buf_idx_0] = reg_lineitem__l_returnflag[ITEM];
KEY_0lineitem__l_linestatus[buf_idx_0] = reg_lineitem__l_linestatus[ITEM];
}
}
__global__ void count_3(size_t COUNT0, uint64_t* COUNT2) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT2, 1);
}
}
__global__ void main_3(size_t COUNT0, DBDecimalType* MAT2aggr0__tmp_attr0, DBDecimalType* MAT2aggr0__tmp_attr1, DBDecimalType* MAT2aggr0__tmp_attr2, DBDecimalType* MAT2aggr0__tmp_attr4, DBDecimalType* MAT2aggr0__tmp_attr6, DBDecimalType* MAT2aggr0__tmp_attr7, DBDecimalType* MAT2aggr0__tmp_attr8, DBI64Type* MAT2aggr0__tmp_attr9, DBCharType* MAT2lineitem__l_linestatus, DBCharType* MAT2lineitem__l_returnflag, uint64_t* MAT_IDX2, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr0__tmp_attr1, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, DBI64Type* aggr0__tmp_attr9, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, DBDecimalType* aggr_rw__rw2, DBI64Type* aggr_rw__rw3, DBDecimalType* aggr_rw__rw4, DBI64Type* aggr_rw__rw5, DBCharType* lineitem__l_linestatus, DBCharType* lineitem__l_returnflag) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize buffers
DBCharType reg_lineitem__l_returnflag[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_lineitem__l_returnflag[ITEM] = lineitem__l_returnflag[ITEM*TB + tid];
}
DBCharType reg_lineitem__l_linestatus[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_lineitem__l_linestatus[ITEM] = lineitem__l_linestatus[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_aggr0__tmp_attr1[ITEM] = aggr0__tmp_attr1[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_aggr0__tmp_attr2[ITEM] = aggr0__tmp_attr2[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_aggr0__tmp_attr4[ITEM] = aggr0__tmp_attr4[ITEM*TB + tid];
}
DBI64Type reg_aggr_rw__rw5[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_aggr_rw__rw5[ITEM] = aggr_rw__rw5[ITEM*TB + tid];
}
DBDecimalType reg_aggr_rw__rw4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_aggr_rw__rw4[ITEM] = aggr_rw__rw4[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr0__tmp_attr6[ITEM] = (reg_aggr_rw__rw4[ITEM]) / ((DBDecimalType)(reg_aggr_rw__rw5[ITEM]));
}
DBI64Type reg_aggr_rw__rw3[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_aggr_rw__rw3[ITEM] = aggr_rw__rw3[ITEM*TB + tid];
}
DBDecimalType reg_aggr_rw__rw2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_aggr_rw__rw2[ITEM] = aggr_rw__rw2[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr7[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr0__tmp_attr7[ITEM] = (reg_aggr_rw__rw2[ITEM]) / ((DBDecimalType)(reg_aggr_rw__rw3[ITEM]));
}
DBI64Type reg_aggr_rw__rw1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_aggr_rw__rw1[ITEM] = aggr_rw__rw1[ITEM*TB + tid];
}
DBDecimalType reg_aggr_rw__rw0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_aggr_rw__rw0[ITEM] = aggr_rw__rw0[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr8[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr0__tmp_attr8[ITEM] = (reg_aggr_rw__rw0[ITEM]) / ((DBDecimalType)(reg_aggr_rw__rw1[ITEM]));
}
DBI64Type reg_aggr0__tmp_attr9[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_aggr0__tmp_attr9[ITEM] = aggr0__tmp_attr9[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx2 = atomicAdd((int*)MAT_IDX2, 1);
MAT2lineitem__l_returnflag[mat_idx2] = reg_lineitem__l_returnflag[ITEM];
MAT2lineitem__l_linestatus[mat_idx2] = reg_lineitem__l_linestatus[ITEM];
MAT2aggr0__tmp_attr0[mat_idx2] = reg_aggr0__tmp_attr0[ITEM];
MAT2aggr0__tmp_attr1[mat_idx2] = reg_aggr0__tmp_attr1[ITEM];
MAT2aggr0__tmp_attr2[mat_idx2] = reg_aggr0__tmp_attr2[ITEM];
MAT2aggr0__tmp_attr4[mat_idx2] = reg_aggr0__tmp_attr4[ITEM];
MAT2aggr0__tmp_attr6[mat_idx2] = reg_aggr0__tmp_attr6[ITEM];
MAT2aggr0__tmp_attr7[mat_idx2] = reg_aggr0__tmp_attr7[ITEM];
MAT2aggr0__tmp_attr8[mat_idx2] = reg_aggr0__tmp_attr8[ITEM];
MAT2aggr0__tmp_attr9[mat_idx2] = reg_aggr0__tmp_attr9[ITEM];
}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto start = std::chrono::high_resolution_clock::now();
//Create aggregation hash table
auto d_HT_0 = cuco::static_map{ (int)5930889*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_1<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_HT_0.ref(cuco::insert), d_lineitem__l_linestatus, d_lineitem__l_returnflag, d_lineitem__l_shipdate, lineitem_size);
size_t COUNT0 = d_HT_0.size();
thrust::device_vector<int64_t> keys_0(COUNT0), vals_0(COUNT0);
d_HT_0.retrieve_all(keys_0.begin(), vals_0.begin());
d_HT_0.clear();
int64_t* raw_keys0 = thrust::raw_pointer_cast(keys_0.data());
insertKeys<<<std::ceil((float)COUNT0/128.), 128>>>(raw_keys0, d_HT_0.ref(cuco::insert), COUNT0);
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr9;
hipMalloc(&d_aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT0);
hipMemset(d_aggr0__tmp_attr9, 0, sizeof(DBI64Type) * COUNT0);
DBDecimalType* d_aggr0__tmp_attr4;
hipMalloc(&d_aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr0__tmp_attr4, 0, sizeof(DBDecimalType) * COUNT0);
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT0);
DBDecimalType* d_aggr0__tmp_attr1;
hipMalloc(&d_aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr0__tmp_attr1, 0, sizeof(DBDecimalType) * COUNT0);
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT0);
DBDecimalType* d_aggr_rw__rw0;
hipMalloc(&d_aggr_rw__rw0, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr_rw__rw0, 0, sizeof(DBDecimalType) * COUNT0);
DBI64Type* d_aggr_rw__rw1;
hipMalloc(&d_aggr_rw__rw1, sizeof(DBI64Type) * COUNT0);
hipMemset(d_aggr_rw__rw1, 0, sizeof(DBI64Type) * COUNT0);
DBDecimalType* d_aggr_rw__rw2;
hipMalloc(&d_aggr_rw__rw2, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr_rw__rw2, 0, sizeof(DBDecimalType) * COUNT0);
DBI64Type* d_aggr_rw__rw3;
hipMalloc(&d_aggr_rw__rw3, sizeof(DBI64Type) * COUNT0);
hipMemset(d_aggr_rw__rw3, 0, sizeof(DBI64Type) * COUNT0);
DBDecimalType* d_aggr_rw__rw4;
hipMalloc(&d_aggr_rw__rw4, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr_rw__rw4, 0, sizeof(DBDecimalType) * COUNT0);
DBI64Type* d_aggr_rw__rw5;
hipMalloc(&d_aggr_rw__rw5, sizeof(DBI64Type) * COUNT0);
hipMemset(d_aggr_rw__rw5, 0, sizeof(DBI64Type) * COUNT0);
DBCharType* d_KEY_0lineitem__l_returnflag;
hipMalloc(&d_KEY_0lineitem__l_returnflag, sizeof(DBCharType) * COUNT0);
hipMemset(d_KEY_0lineitem__l_returnflag, 0, sizeof(DBCharType) * COUNT0);
DBCharType* d_KEY_0lineitem__l_linestatus;
hipMalloc(&d_KEY_0lineitem__l_linestatus, sizeof(DBCharType) * COUNT0);
hipMemset(d_KEY_0lineitem__l_linestatus, 0, sizeof(DBCharType) * COUNT0);
main_1<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_HT_0.ref(cuco::find), d_KEY_0lineitem__l_linestatus, d_KEY_0lineitem__l_returnflag, d_aggr0__tmp_attr0, d_aggr0__tmp_attr1, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_aggr0__tmp_attr9, d_aggr_rw__rw0, d_aggr_rw__rw1, d_aggr_rw__rw2, d_aggr_rw__rw3, d_aggr_rw__rw4, d_aggr_rw__rw5, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_linestatus, d_lineitem__l_quantity, d_lineitem__l_returnflag, d_lineitem__l_shipdate, d_lineitem__l_tax, lineitem_size);
//Materialize count
uint64_t* d_COUNT2;
hipMalloc(&d_COUNT2, sizeof(uint64_t));
hipMemset(d_COUNT2, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)COUNT0/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT0, d_COUNT2);
uint64_t COUNT2;
hipMemcpy(&COUNT2, d_COUNT2, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX2;
hipMalloc(&d_MAT_IDX2, sizeof(uint64_t));
hipMemset(d_MAT_IDX2, 0, sizeof(uint64_t));
auto MAT2lineitem__l_returnflag = (DBCharType*)malloc(sizeof(DBCharType) * COUNT2);
DBCharType* d_MAT2lineitem__l_returnflag;
hipMalloc(&d_MAT2lineitem__l_returnflag, sizeof(DBCharType) * COUNT2);
auto MAT2lineitem__l_linestatus = (DBCharType*)malloc(sizeof(DBCharType) * COUNT2);
DBCharType* d_MAT2lineitem__l_linestatus;
hipMalloc(&d_MAT2lineitem__l_linestatus, sizeof(DBCharType) * COUNT2);
auto MAT2aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT2);
DBDecimalType* d_MAT2aggr0__tmp_attr0;
hipMalloc(&d_MAT2aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT2);
auto MAT2aggr0__tmp_attr1 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT2);
DBDecimalType* d_MAT2aggr0__tmp_attr1;
hipMalloc(&d_MAT2aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT2);
auto MAT2aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT2);
DBDecimalType* d_MAT2aggr0__tmp_attr2;
hipMalloc(&d_MAT2aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT2);
auto MAT2aggr0__tmp_attr4 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT2);
DBDecimalType* d_MAT2aggr0__tmp_attr4;
hipMalloc(&d_MAT2aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT2);
auto MAT2aggr0__tmp_attr6 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT2);
DBDecimalType* d_MAT2aggr0__tmp_attr6;
hipMalloc(&d_MAT2aggr0__tmp_attr6, sizeof(DBDecimalType) * COUNT2);
auto MAT2aggr0__tmp_attr7 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT2);
DBDecimalType* d_MAT2aggr0__tmp_attr7;
hipMalloc(&d_MAT2aggr0__tmp_attr7, sizeof(DBDecimalType) * COUNT2);
auto MAT2aggr0__tmp_attr8 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT2);
DBDecimalType* d_MAT2aggr0__tmp_attr8;
hipMalloc(&d_MAT2aggr0__tmp_attr8, sizeof(DBDecimalType) * COUNT2);
auto MAT2aggr0__tmp_attr9 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT2);
DBI64Type* d_MAT2aggr0__tmp_attr9;
hipMalloc(&d_MAT2aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT2);
main_3<<<std::ceil((float)COUNT0/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT0, d_MAT2aggr0__tmp_attr0, d_MAT2aggr0__tmp_attr1, d_MAT2aggr0__tmp_attr2, d_MAT2aggr0__tmp_attr4, d_MAT2aggr0__tmp_attr6, d_MAT2aggr0__tmp_attr7, d_MAT2aggr0__tmp_attr8, d_MAT2aggr0__tmp_attr9, d_MAT2lineitem__l_linestatus, d_MAT2lineitem__l_returnflag, d_MAT_IDX2, d_aggr0__tmp_attr0, d_aggr0__tmp_attr1, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_aggr0__tmp_attr9, d_aggr_rw__rw0, d_aggr_rw__rw1, d_aggr_rw__rw2, d_aggr_rw__rw3, d_aggr_rw__rw4, d_aggr_rw__rw5, d_KEY_0lineitem__l_linestatus, d_KEY_0lineitem__l_returnflag);
hipMemcpy(MAT2lineitem__l_returnflag, d_MAT2lineitem__l_returnflag, sizeof(DBCharType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2lineitem__l_linestatus, d_MAT2lineitem__l_linestatus, sizeof(DBCharType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2aggr0__tmp_attr0, d_MAT2aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2aggr0__tmp_attr1, d_MAT2aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2aggr0__tmp_attr2, d_MAT2aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2aggr0__tmp_attr4, d_MAT2aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2aggr0__tmp_attr6, d_MAT2aggr0__tmp_attr6, sizeof(DBDecimalType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2aggr0__tmp_attr7, d_MAT2aggr0__tmp_attr7, sizeof(DBDecimalType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2aggr0__tmp_attr8, d_MAT2aggr0__tmp_attr8, sizeof(DBDecimalType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2aggr0__tmp_attr9, d_MAT2aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT2, hipMemcpyDeviceToHost);
auto end = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < COUNT2; i++) { std::cout << "" << MAT2lineitem__l_returnflag[i];
std::cout << "|" << MAT2lineitem__l_linestatus[i];
std::cout << "|" << MAT2aggr0__tmp_attr0[i];
std::cout << "|" << MAT2aggr0__tmp_attr1[i];
std::cout << "|" << MAT2aggr0__tmp_attr2[i];
std::cout << "|" << MAT2aggr0__tmp_attr4[i];
std::cout << "|" << MAT2aggr0__tmp_attr6[i];
std::cout << "|" << MAT2aggr0__tmp_attr7[i];
std::cout << "|" << MAT2aggr0__tmp_attr8[i];
std::cout << "|" << MAT2aggr0__tmp_attr9[i];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
      size_t aux_mem = usedGpuMem() - used_mem;
      std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_KEY_0lineitem__l_linestatus);
hipFree(d_KEY_0lineitem__l_returnflag);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr0__tmp_attr1);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_aggr0__tmp_attr4);
hipFree(d_aggr0__tmp_attr9);
hipFree(d_aggr_rw__rw0);
hipFree(d_aggr_rw__rw1);
hipFree(d_aggr_rw__rw2);
hipFree(d_aggr_rw__rw3);
hipFree(d_aggr_rw__rw4);
hipFree(d_aggr_rw__rw5);
hipFree(d_COUNT2);
hipFree(d_MAT2aggr0__tmp_attr0);
hipFree(d_MAT2aggr0__tmp_attr1);
hipFree(d_MAT2aggr0__tmp_attr2);
hipFree(d_MAT2aggr0__tmp_attr4);
hipFree(d_MAT2aggr0__tmp_attr6);
hipFree(d_MAT2aggr0__tmp_attr7);
hipFree(d_MAT2aggr0__tmp_attr8);
hipFree(d_MAT2aggr0__tmp_attr9);
hipFree(d_MAT2lineitem__l_linestatus);
hipFree(d_MAT2lineitem__l_returnflag);
hipFree(d_MAT_IDX2);
free(MAT2aggr0__tmp_attr0);
free(MAT2aggr0__tmp_attr1);
free(MAT2aggr0__tmp_attr2);
free(MAT2aggr0__tmp_attr4);
free(MAT2aggr0__tmp_attr6);
free(MAT2aggr0__tmp_attr7);
free(MAT2aggr0__tmp_attr8);
free(MAT2aggr0__tmp_attr9);
free(MAT2lineitem__l_linestatus);
free(MAT2lineitem__l_returnflag);
}