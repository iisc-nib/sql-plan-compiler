#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_1(uint64_t* COUNT0, DBStringType* part__p_brand, DBI32Type* part__p_size, DBStringType* part__p_type, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_size = part__p_size[tid];
if (!((evaluatePredicate(reg_part__p_size, 49, Predicate::eq)) || (evaluatePredicate(reg_part__p_size, 14, Predicate::eq)) || (evaluatePredicate(reg_part__p_size, 23, Predicate::eq)) || (evaluatePredicate(reg_part__p_size, 45, Predicate::eq)) || (evaluatePredicate(reg_part__p_size, 19, Predicate::eq)) || (evaluatePredicate(reg_part__p_size, 3, Predicate::eq)) || (evaluatePredicate(reg_part__p_size, 36, Predicate::eq)) || (evaluatePredicate(reg_part__p_size, 9, Predicate::eq)))) return;
auto reg_part__p_brand = part__p_brand[tid];
if (!(evaluatePredicate(reg_part__p_brand, "Brand#45", Predicate::neq))) return;
auto reg_part__p_type = part__p_type[tid];
if (!(!(Like(reg_part__p_type, "MEDIUM POLISHED", "", nullptr, nullptr, 0)))) return;
//Materialize count
atomicAdd((int*)COUNT0, 1);
}
template<typename HASHTABLE_INSERT_PK>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT_PK HT_0, DBStringType* part__p_brand, DBI32Type* part__p_partkey, DBI32Type* part__p_size, DBStringType* part__p_type, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_size = part__p_size[tid];
if (!((evaluatePredicate(reg_part__p_size, 49, Predicate::eq)) || (evaluatePredicate(reg_part__p_size, 14, Predicate::eq)) || (evaluatePredicate(reg_part__p_size, 23, Predicate::eq)) || (evaluatePredicate(reg_part__p_size, 45, Predicate::eq)) || (evaluatePredicate(reg_part__p_size, 19, Predicate::eq)) || (evaluatePredicate(reg_part__p_size, 3, Predicate::eq)) || (evaluatePredicate(reg_part__p_size, 36, Predicate::eq)) || (evaluatePredicate(reg_part__p_size, 9, Predicate::eq)))) return;
auto reg_part__p_brand = part__p_brand[tid];
if (!(evaluatePredicate(reg_part__p_brand, "Brand#45", Predicate::neq))) return;
auto reg_part__p_type = part__p_type[tid];
if (!(!(Like(reg_part__p_type, "MEDIUM POLISHED", "", nullptr, nullptr, 0)))) return;
uint64_t KEY_0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_0 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0, buf_idx_0});
BUF_0[buf_idx_0 * 1 + 0] = tid;
}
__global__ void count_3(uint64_t* COUNT2, DBStringType* supplier__s_comment, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_comment = supplier__s_comment[tid];
if (!(Like(reg_supplier__s_comment, "", "", (const char*[]){ "Customer", "Complaints" }, (const int[]){ 8, 10 }, 2))) return;
//Materialize count
atomicAdd((int*)COUNT2, 1);
}
template<typename HASHTABLE_INSERT_SJ>
__global__ void main_3(HASHTABLE_INSERT_SJ HT_2, DBStringType* supplier__s_comment, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_comment = supplier__s_comment[tid];
if (!(Like(reg_supplier__s_comment, "", "", (const char*[]){ "Customer", "Complaints" }, (const int[]){ 8, 10 }, 2))) return;
uint64_t KEY_2 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_2 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
HT_2.insert(cuco::pair{KEY_2, 1});
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_PROBE_SJ, typename HASHTABLE_INSERT>
__global__ void count_5(uint64_t* BUF_0, HASHTABLE_PROBE_PK HT_0, HASHTABLE_PROBE_SJ HT_2, HASHTABLE_INSERT HT_4, DBI16Type* part__p_brand_encoded, DBI32Type* part__p_size, DBI16Type* part__p_type_encoded, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= partsupp_size) return;
uint64_t KEY_0 = 0;
auto reg_partsupp__ps_partkey = partsupp__ps_partkey[tid];

KEY_0 |= reg_partsupp__ps_partkey;
//Probe Hash table
auto SLOT_0 = HT_0.find(KEY_0);
if (SLOT_0 == HT_0.end()) return;
if (!(true)) return;
uint64_t KEY_2 = 0;
auto reg_partsupp__ps_suppkey = partsupp__ps_suppkey[tid];

KEY_2 |= reg_partsupp__ps_suppkey;
//Probe Hash table
auto SLOT_2 = HT_2.find(KEY_2);
if (!(SLOT_2 == HT_2.end())) return;
if (!(true)) return;
uint64_t KEY_4 = 0;
auto reg_part__p_brand_encoded = part__p_brand_encoded[BUF_0[SLOT_0->second * 1 + 0]];

KEY_4 |= reg_part__p_brand_encoded;
auto reg_part__p_type_encoded = part__p_type_encoded[BUF_0[SLOT_0->second * 1 + 0]];
KEY_4 <<= 16;
KEY_4 |= reg_part__p_type_encoded;
auto reg_part__p_size = part__p_size[BUF_0[SLOT_0->second * 1 + 0]];
KEY_4 <<= 32;
KEY_4 |= reg_part__p_size;
//Create aggregation hash table
HT_4.insert(cuco::pair{KEY_4, 1});
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_PROBE_SJ, typename HASHTABLE_FIND>
__global__ void main_5(uint64_t* BUF_0, HASHTABLE_PROBE_PK HT_0, HASHTABLE_PROBE_SJ HT_2, HASHTABLE_FIND HT_4, DBI16Type* KEY_4part__p_brand_encoded, DBI32Type* KEY_4part__p_size, DBI16Type* KEY_4part__p_type_encoded, DBI64Type* aggr0__tmp_attr0, DBI16Type* part__p_brand_encoded, DBI32Type* part__p_size, DBI16Type* part__p_type_encoded, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= partsupp_size) return;
uint64_t KEY_0 = 0;
auto reg_partsupp__ps_partkey = partsupp__ps_partkey[tid];

KEY_0 |= reg_partsupp__ps_partkey;
//Probe Hash table
auto SLOT_0 = HT_0.find(KEY_0);
if (SLOT_0 == HT_0.end()) return;
if (!(true)) return;
uint64_t KEY_2 = 0;
auto reg_partsupp__ps_suppkey = partsupp__ps_suppkey[tid];

KEY_2 |= reg_partsupp__ps_suppkey;
//Probe Hash table
auto SLOT_2 = HT_2.find(KEY_2);
if (!(SLOT_2 == HT_2.end())) return;
if (!(true)) return;
uint64_t KEY_4 = 0;
auto reg_part__p_brand_encoded = part__p_brand_encoded[BUF_0[SLOT_0->second * 1 + 0]];

KEY_4 |= reg_part__p_brand_encoded;
auto reg_part__p_type_encoded = part__p_type_encoded[BUF_0[SLOT_0->second * 1 + 0]];
KEY_4 <<= 16;
KEY_4 |= reg_part__p_type_encoded;
auto reg_part__p_size = part__p_size[BUF_0[SLOT_0->second * 1 + 0]];
KEY_4 <<= 32;
KEY_4 |= reg_part__p_size;
//Aggregate in hashtable
auto buf_idx_4 = HT_4.find(KEY_4)->second;
aggregate_sum(&aggr0__tmp_attr0[buf_idx_4], 1);
KEY_4part__p_brand_encoded[buf_idx_4] = reg_part__p_brand_encoded;
KEY_4part__p_type_encoded[buf_idx_4] = reg_part__p_type_encoded;
KEY_4part__p_size[buf_idx_4] = reg_part__p_size;
}
__global__ void count_7(size_t COUNT4, uint64_t* COUNT6) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT4) return;
//Materialize count
atomicAdd((int*)COUNT6, 1);
}
__global__ void main_7(size_t COUNT4, DBI64Type* MAT6aggr0__tmp_attr0, DBI16Type* MAT6part__p_brand_encoded, DBI32Type* MAT6part__p_size, DBI16Type* MAT6part__p_type_encoded, uint64_t* MAT_IDX6, DBI64Type* aggr0__tmp_attr0, DBI16Type* part__p_brand_encoded, DBI32Type* part__p_size, DBI16Type* part__p_type_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT4) return;
//Materialize buffers
auto mat_idx6 = atomicAdd((int*)MAT_IDX6, 1);
auto reg_part__p_brand_encoded = part__p_brand_encoded[tid];
MAT6part__p_brand_encoded[mat_idx6] = reg_part__p_brand_encoded;
auto reg_part__p_type_encoded = part__p_type_encoded[tid];
MAT6part__p_type_encoded[mat_idx6] = reg_part__p_type_encoded;
auto reg_part__p_size = part__p_size[tid];
MAT6part__p_size[mat_idx6] = reg_part__p_size;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT6aggr0__tmp_attr0[mat_idx6] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)part_size/128.), 128>>>(d_COUNT0, d_part__p_brand, d_part__p_size, d_part__p_type, part_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_part__p_brand, d_part__p_partkey, d_part__p_size, d_part__p_type, part_size);
//Materialize count
uint64_t* d_COUNT2;
hipMalloc(&d_COUNT2, sizeof(uint64_t));
hipMemset(d_COUNT2, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT2, d_supplier__s_comment, supplier_size);
uint64_t COUNT2;
hipMemcpy(&COUNT2, d_COUNT2, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)supplier_size/128.), 128>>>(d_HT_2.ref(cuco::insert), d_supplier__s_comment, d_supplier__s_suppkey, supplier_size);
//Create aggregation hash table
auto d_HT_4 = cuco::static_map{ (int)120976*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5<<<std::ceil((float)partsupp_size/128.), 128>>>(d_BUF_0, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::insert), d_part__p_brand_encoded, d_part__p_size, d_part__p_type_encoded, d_partsupp__ps_partkey, d_partsupp__ps_suppkey, partsupp_size);
size_t COUNT4 = d_HT_4.size();
thrust::device_vector<int64_t> keys_4(COUNT4), vals_4(COUNT4);
d_HT_4.retrieve_all(keys_4.begin(), vals_4.begin());
d_HT_4.clear();
int64_t* raw_keys4 = thrust::raw_pointer_cast(keys_4.data());
insertKeys<<<std::ceil((float)COUNT4/128.), 128>>>(raw_keys4, d_HT_4.ref(cuco::insert), COUNT4);
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT4);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT4);
DBI16Type* d_KEY_4part__p_brand_encoded;
hipMalloc(&d_KEY_4part__p_brand_encoded, sizeof(DBI16Type) * COUNT4);
hipMemset(d_KEY_4part__p_brand_encoded, 0, sizeof(DBI16Type) * COUNT4);
DBI16Type* d_KEY_4part__p_type_encoded;
hipMalloc(&d_KEY_4part__p_type_encoded, sizeof(DBI16Type) * COUNT4);
hipMemset(d_KEY_4part__p_type_encoded, 0, sizeof(DBI16Type) * COUNT4);
DBI32Type* d_KEY_4part__p_size;
hipMalloc(&d_KEY_4part__p_size, sizeof(DBI32Type) * COUNT4);
hipMemset(d_KEY_4part__p_size, 0, sizeof(DBI32Type) * COUNT4);
main_5<<<std::ceil((float)partsupp_size/128.), 128>>>(d_BUF_0, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::find), d_KEY_4part__p_brand_encoded, d_KEY_4part__p_size, d_KEY_4part__p_type_encoded, d_aggr0__tmp_attr0, d_part__p_brand_encoded, d_part__p_size, d_part__p_type_encoded, d_partsupp__ps_partkey, d_partsupp__ps_suppkey, partsupp_size);
//Materialize count
uint64_t* d_COUNT6;
hipMalloc(&d_COUNT6, sizeof(uint64_t));
hipMemset(d_COUNT6, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)COUNT4/128.), 128>>>(COUNT4, d_COUNT6);
uint64_t COUNT6;
hipMemcpy(&COUNT6, d_COUNT6, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX6;
hipMalloc(&d_MAT_IDX6, sizeof(uint64_t));
hipMemset(d_MAT_IDX6, 0, sizeof(uint64_t));
auto MAT6part__p_brand_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT6);
DBI16Type* d_MAT6part__p_brand_encoded;
hipMalloc(&d_MAT6part__p_brand_encoded, sizeof(DBI16Type) * COUNT6);
auto MAT6part__p_type_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT6);
DBI16Type* d_MAT6part__p_type_encoded;
hipMalloc(&d_MAT6part__p_type_encoded, sizeof(DBI16Type) * COUNT6);
auto MAT6part__p_size = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT6);
DBI32Type* d_MAT6part__p_size;
hipMalloc(&d_MAT6part__p_size, sizeof(DBI32Type) * COUNT6);
auto MAT6aggr0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT6);
DBI64Type* d_MAT6aggr0__tmp_attr0;
hipMalloc(&d_MAT6aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT6);
main_7<<<std::ceil((float)COUNT4/128.), 128>>>(COUNT4, d_MAT6aggr0__tmp_attr0, d_MAT6part__p_brand_encoded, d_MAT6part__p_size, d_MAT6part__p_type_encoded, d_MAT_IDX6, d_aggr0__tmp_attr0, d_KEY_4part__p_brand_encoded, d_KEY_4part__p_size, d_KEY_4part__p_type_encoded);
hipMemcpy(MAT6part__p_brand_encoded, d_MAT6part__p_brand_encoded, sizeof(DBI16Type) * COUNT6, hipMemcpyDeviceToHost);
hipMemcpy(MAT6part__p_type_encoded, d_MAT6part__p_type_encoded, sizeof(DBI16Type) * COUNT6, hipMemcpyDeviceToHost);
hipMemcpy(MAT6part__p_size, d_MAT6part__p_size, sizeof(DBI32Type) * COUNT6, hipMemcpyDeviceToHost);
hipMemcpy(MAT6aggr0__tmp_attr0, d_MAT6aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT6, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT6; i++) { std::cout << "" << part__p_brand_map[MAT6part__p_brand_encoded[i]];
std::cout << "|" << part__p_type_map[MAT6part__p_type_encoded[i]];
std::cout << "|" << MAT6part__p_size[i];
std::cout << "|" << MAT6aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_COUNT0);
hipFree(d_COUNT2);
hipFree(d_KEY_4part__p_brand_encoded);
hipFree(d_KEY_4part__p_size);
hipFree(d_KEY_4part__p_type_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT6);
hipFree(d_MAT6aggr0__tmp_attr0);
hipFree(d_MAT6part__p_brand_encoded);
hipFree(d_MAT6part__p_size);
hipFree(d_MAT6part__p_type_encoded);
hipFree(d_MAT_IDX6);
free(MAT6aggr0__tmp_attr0);
free(MAT6part__p_brand_encoded);
free(MAT6part__p_size);
free(MAT6part__p_type_encoded);
}