#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
template<typename HASHTABLE_INSERT>
__global__ void count_5977caed0bf0(HASHTABLE_INSERT HT_5977cae885c0, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_quantity, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::lt))) return;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
if (!(evaluatePredicate(reg_lineitem__l_discount, 0.05, Predicate::gte) && evaluatePredicate(reg_lineitem__l_discount, 0.07, Predicate::lte))) return;
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
if (!(evaluatePredicate(reg_lineitem__l_quantity, 24.0, Predicate::lt))) return;
uint64_t KEY_5977cae885c0 = 0;
//Create aggregation hash table
HT_5977cae885c0.insert(cuco::pair{KEY_5977cae885c0, 1});
}
template<typename HASHTABLE_FIND>
__global__ void main_5977caed0bf0(HASHTABLE_FIND HT_5977cae885c0, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBDecimalType* lineitem__l_quantity, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::lt))) return;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
if (!(evaluatePredicate(reg_lineitem__l_discount, 0.05, Predicate::gte) && evaluatePredicate(reg_lineitem__l_discount, 0.07, Predicate::lte))) return;
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
if (!(evaluatePredicate(reg_lineitem__l_quantity, 24.0, Predicate::lt))) return;
uint64_t KEY_5977cae885c0 = 0;
//Aggregate in hashtable
auto buf_idx_5977cae885c0 = HT_5977cae885c0.find(KEY_5977cae885c0)->second;
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * (reg_lineitem__l_discount);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5977cae885c0], reg_map0__tmp_attr1);
}
__global__ void count_5977caed2820(size_t COUNT5977cae885c0, uint64_t* COUNT5977cae9b770) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5977cae885c0) return;
//Materialize count
atomicAdd((int*)COUNT5977cae9b770, 1);
}
__global__ void main_5977caed2820(size_t COUNT5977cae885c0, DBDecimalType* MAT5977cae9b770aggr0__tmp_attr0, uint64_t* MAT_IDX5977cae9b770, DBDecimalType* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5977cae885c0) return;
//Materialize buffers
auto mat_idx5977cae9b770 = atomicAdd((int*)MAT_IDX5977cae9b770, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5977cae9b770aggr0__tmp_attr0[mat_idx5977cae9b770] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map) {
//Create aggregation hash table
auto d_HT_5977cae885c0 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5977caed0bf0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_HT_5977cae885c0.ref(cuco::insert), d_lineitem__l_discount, d_lineitem__l_quantity, d_lineitem__l_shipdate, lineitem_size);
size_t COUNT5977cae885c0 = d_HT_5977cae885c0.size();
thrust::device_vector<int64_t> keys_5977cae885c0(COUNT5977cae885c0), vals_5977cae885c0(COUNT5977cae885c0);
d_HT_5977cae885c0.retrieve_all(keys_5977cae885c0.begin(), vals_5977cae885c0.begin());
d_HT_5977cae885c0.clear();
int64_t* raw_keys5977cae885c0 = thrust::raw_pointer_cast(keys_5977cae885c0.data());
insertKeys<<<std::ceil((float)COUNT5977cae885c0/32.), 32>>>(raw_keys5977cae885c0, d_HT_5977cae885c0.ref(cuco::insert), COUNT5977cae885c0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5977cae885c0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5977cae885c0);
main_5977caed0bf0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_HT_5977cae885c0.ref(cuco::find), d_aggr0__tmp_attr0, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_quantity, d_lineitem__l_shipdate, lineitem_size);
//Materialize count
uint64_t* d_COUNT5977cae9b770;
hipMalloc(&d_COUNT5977cae9b770, sizeof(uint64_t));
hipMemset(d_COUNT5977cae9b770, 0, sizeof(uint64_t));
count_5977caed2820<<<std::ceil((float)COUNT5977cae885c0/32.), 32>>>(COUNT5977cae885c0, d_COUNT5977cae9b770);
uint64_t COUNT5977cae9b770;
hipMemcpy(&COUNT5977cae9b770, d_COUNT5977cae9b770, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5977cae9b770;
hipMalloc(&d_MAT_IDX5977cae9b770, sizeof(uint64_t));
hipMemset(d_MAT_IDX5977cae9b770, 0, sizeof(uint64_t));
auto MAT5977cae9b770aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5977cae9b770);
DBDecimalType* d_MAT5977cae9b770aggr0__tmp_attr0;
hipMalloc(&d_MAT5977cae9b770aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5977cae9b770);
main_5977caed2820<<<std::ceil((float)COUNT5977cae885c0/32.), 32>>>(COUNT5977cae885c0, d_MAT5977cae9b770aggr0__tmp_attr0, d_MAT_IDX5977cae9b770, d_aggr0__tmp_attr0);
hipMemcpy(MAT5977cae9b770aggr0__tmp_attr0, d_MAT5977cae9b770aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5977cae9b770, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5977cae9b770; i++) { std::cout << MAT5977cae9b770aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5977cae9b770);
hipFree(d_MAT5977cae9b770aggr0__tmp_attr0);
hipFree(d_MAT_IDX5977cae9b770);
free(MAT5977cae9b770aggr0__tmp_attr0);
}