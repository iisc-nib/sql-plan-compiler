#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
template<typename HASHTABLE_INSERT>
__global__ void count_1(HASHTABLE_INSERT HT_0, size_t lineitem_size, DBI32Type* lineitem_u_1__l_orderkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_0 = 0;
auto reg_lineitem_u_1__l_orderkey = lineitem_u_1__l_orderkey[tid];

KEY_0 |= reg_lineitem_u_1__l_orderkey;
//Create aggregation hash table
HT_0.insert(cuco::pair{KEY_0, 1});
}
template<typename HASHTABLE_FIND>
__global__ void main_1(HASHTABLE_FIND HT_0, DBI32Type* KEY_0lineitem_u_1__l_orderkey, DBDecimalType* aggr0__tmp_attr0, size_t lineitem_size, DBI32Type* lineitem_u_1__l_orderkey, DBDecimalType* lineitem_u_1__l_quantity) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_0 = 0;
auto reg_lineitem_u_1__l_orderkey = lineitem_u_1__l_orderkey[tid];

KEY_0 |= reg_lineitem_u_1__l_orderkey;
//Aggregate in hashtable
auto buf_idx_0 = HT_0.find(KEY_0)->second;
auto reg_lineitem_u_1__l_quantity = lineitem_u_1__l_quantity[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_0], reg_lineitem_u_1__l_quantity);
KEY_0lineitem_u_1__l_orderkey[buf_idx_0] = reg_lineitem_u_1__l_orderkey;
}
__global__ void count_3(size_t COUNT0, uint64_t* COUNT2, DBDecimalType* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT0) return;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
if (!(evaluatePredicate(reg_aggr0__tmp_attr0, 300.0, Predicate::gt))) return;
//Materialize count
atomicAdd((int*)COUNT2, 1);
}
template<typename HASHTABLE_INSERT_SJ>
__global__ void main_3(size_t COUNT0, HASHTABLE_INSERT_SJ HT_2, DBDecimalType* aggr0__tmp_attr0, DBI32Type* lineitem_u_1__l_orderkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT0) return;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
if (!(evaluatePredicate(reg_aggr0__tmp_attr0, 300.0, Predicate::gt))) return;
uint64_t KEY_2 = 0;
auto reg_lineitem_u_1__l_orderkey = lineitem_u_1__l_orderkey[tid];

KEY_2 |= reg_lineitem_u_1__l_orderkey;
// Insert hash table kernel;
HT_2.insert(cuco::pair{KEY_2, 1});
}
__global__ void count_5(uint64_t* COUNT4, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
//Materialize count
atomicAdd((int*)COUNT4, 1);
}
template<typename HASHTABLE_INSERT_PK>
__global__ void main_5(uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_INSERT_PK HT_4, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_4 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_4 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_4 = atomicAdd((int*)BUF_IDX_4, 1);
HT_4.insert(cuco::pair{KEY_4, buf_idx_4});
BUF_4[buf_idx_4 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE_SJ, typename HASHTABLE_PROBE_PK>
__global__ void count_7(uint64_t* BUF_4, uint64_t* COUNT6, HASHTABLE_PROBE_SJ HT_2, HASHTABLE_PROBE_PK HT_4, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_2 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_2 |= reg_orders__o_orderkey;
//Probe Hash table
auto SLOT_2 = HT_2.find(KEY_2);
if (SLOT_2 == HT_2.end()) return;
if (!(true)) return;
uint64_t KEY_4 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_4 |= reg_orders__o_custkey;
//Probe Hash table
auto SLOT_4 = HT_4.find(KEY_4);
if (SLOT_4 == HT_4.end()) return;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT6, 1);
}
template<typename HASHTABLE_PROBE_SJ, typename HASHTABLE_PROBE_PK, typename HASHTABLE_INSERT_PK>
__global__ void main_7(uint64_t* BUF_4, uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_PROBE_SJ HT_2, HASHTABLE_PROBE_PK HT_4, HASHTABLE_INSERT_PK HT_6, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_2 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_2 |= reg_orders__o_orderkey;
//Probe Hash table
auto SLOT_2 = HT_2.find(KEY_2);
if (SLOT_2 == HT_2.end()) return;
if (!(true)) return;
uint64_t KEY_4 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_4 |= reg_orders__o_custkey;
//Probe Hash table
auto SLOT_4 = HT_4.find(KEY_4);
if (SLOT_4 == HT_4.end()) return;
if (!(true)) return;
uint64_t KEY_6 = 0;

KEY_6 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_6 = atomicAdd((int*)BUF_IDX_6, 1);
HT_6.insert(cuco::pair{KEY_6, buf_idx_6});
BUF_6[buf_idx_6 * 2 + 0] = BUF_4[SLOT_4->second * 1 + 0];
BUF_6[buf_idx_6 * 2 + 1] = tid;
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_INSERT>
__global__ void count_9(uint64_t* BUF_6, HASHTABLE_PROBE_PK HT_6, HASHTABLE_INSERT HT_8, DBI32Type* lineitem__l_orderkey, size_t lineitem_size, DBI32Type* orders__o_orderkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_6 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_6 |= reg_lineitem__l_orderkey;
//Probe Hash table
auto SLOT_6 = HT_6.find(KEY_6);
if (SLOT_6 == HT_6.end()) return;
if (!(true)) return;
uint64_t KEY_8 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[BUF_6[SLOT_6->second * 2 + 1]];

KEY_8 |= reg_orders__o_orderkey;
//Create aggregation hash table
HT_8.insert(cuco::pair{KEY_8, 1});
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_FIND>
__global__ void main_9(uint64_t* BUF_6, HASHTABLE_PROBE_PK HT_6, HASHTABLE_FIND HT_8, DBI32Type* KEY_8orders__o_orderkey, DBDecimalType* aggr1__tmp_attr1, DBI32Type* aggr__c_custkey, DBDateType* aggr__o_orderdate, DBDecimalType* aggr__o_totalprice, DBI32Type* customer__c_custkey, DBI32Type* lineitem__l_orderkey, DBDecimalType* lineitem__l_quantity, size_t lineitem_size, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, DBDecimalType* orders__o_totalprice) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_6 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_6 |= reg_lineitem__l_orderkey;
//Probe Hash table
auto SLOT_6 = HT_6.find(KEY_6);
if (SLOT_6 == HT_6.end()) return;
if (!(true)) return;
uint64_t KEY_8 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[BUF_6[SLOT_6->second * 2 + 1]];

KEY_8 |= reg_orders__o_orderkey;
//Aggregate in hashtable
auto buf_idx_8 = HT_8.find(KEY_8)->second;
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
aggregate_sum(&aggr1__tmp_attr1[buf_idx_8], reg_lineitem__l_quantity);
auto reg_customer__c_custkey = customer__c_custkey[BUF_6[SLOT_6->second * 2 + 0]];
aggregate_any(&aggr__c_custkey[buf_idx_8], reg_customer__c_custkey);
auto reg_orders__o_totalprice = orders__o_totalprice[BUF_6[SLOT_6->second * 2 + 1]];
aggregate_any(&aggr__o_totalprice[buf_idx_8], reg_orders__o_totalprice);
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_6[SLOT_6->second * 2 + 1]];
aggregate_any(&aggr__o_orderdate[buf_idx_8], reg_orders__o_orderdate);
KEY_8orders__o_orderkey[buf_idx_8] = reg_orders__o_orderkey;
}
__global__ void count_11(uint64_t* COUNT10, size_t COUNT8) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT8) return;
//Materialize count
atomicAdd((int*)COUNT10, 1);
}
__global__ void main_11(size_t COUNT8, DBDecimalType* MAT10aggr1__tmp_attr1, DBI32Type* MAT10aggr__c_custkey, DBDateType* MAT10aggr__o_orderdate, DBDecimalType* MAT10aggr__o_totalprice, DBI32Type* MAT10orders__o_orderkey, uint64_t* MAT_IDX10, DBDecimalType* aggr1__tmp_attr1, DBI32Type* aggr__c_custkey, DBDateType* aggr__o_orderdate, DBDecimalType* aggr__o_totalprice, DBI32Type* orders__o_orderkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT8) return;
//Materialize buffers
auto mat_idx10 = atomicAdd((int*)MAT_IDX10, 1);
auto reg_aggr__c_custkey = aggr__c_custkey[tid];
MAT10aggr__c_custkey[mat_idx10] = reg_aggr__c_custkey;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];
MAT10orders__o_orderkey[mat_idx10] = reg_orders__o_orderkey;
auto reg_aggr__o_orderdate = aggr__o_orderdate[tid];
MAT10aggr__o_orderdate[mat_idx10] = reg_aggr__o_orderdate;
auto reg_aggr__o_totalprice = aggr__o_totalprice[tid];
MAT10aggr__o_totalprice[mat_idx10] = reg_aggr__o_totalprice;
auto reg_aggr1__tmp_attr1 = aggr1__tmp_attr1[tid];
MAT10aggr1__tmp_attr1[mat_idx10] = reg_aggr1__tmp_attr1;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
//Create aggregation hash table
auto d_HT_0 = cuco::static_map{ (int)6001215*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_1<<<std::ceil((float)lineitem_size/128.), 128>>>(d_HT_0.ref(cuco::insert), lineitem_size, d_lineitem__l_orderkey);
size_t COUNT0 = d_HT_0.size();
thrust::device_vector<int64_t> keys_0(COUNT0), vals_0(COUNT0);
d_HT_0.retrieve_all(keys_0.begin(), vals_0.begin());
d_HT_0.clear();
int64_t* raw_keys0 = thrust::raw_pointer_cast(keys_0.data());
insertKeys<<<std::ceil((float)COUNT0/128.), 128>>>(raw_keys0, d_HT_0.ref(cuco::insert), COUNT0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT0);
DBI32Type* d_KEY_0lineitem_u_1__l_orderkey;
hipMalloc(&d_KEY_0lineitem_u_1__l_orderkey, sizeof(DBI32Type) * COUNT0);
hipMemset(d_KEY_0lineitem_u_1__l_orderkey, 0, sizeof(DBI32Type) * COUNT0);
main_1<<<std::ceil((float)lineitem_size/128.), 128>>>(d_HT_0.ref(cuco::find), d_KEY_0lineitem_u_1__l_orderkey, d_aggr0__tmp_attr0, lineitem_size, d_lineitem__l_orderkey, d_lineitem__l_quantity);
//Materialize count
uint64_t* d_COUNT2;
hipMalloc(&d_COUNT2, sizeof(uint64_t));
hipMemset(d_COUNT2, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)COUNT0/128.), 128>>>(COUNT0, d_COUNT2, d_aggr0__tmp_attr0);
uint64_t COUNT2;
hipMemcpy(&COUNT2, d_COUNT2, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)COUNT0/128.), 128>>>(COUNT0, d_HT_2.ref(cuco::insert), d_aggr0__tmp_attr0, d_KEY_0lineitem_u_1__l_orderkey);
//Materialize count
uint64_t* d_COUNT4;
hipMalloc(&d_COUNT4, sizeof(uint64_t));
hipMemset(d_COUNT4, 0, sizeof(uint64_t));
count_5<<<std::ceil((float)customer_size/128.), 128>>>(d_COUNT4, customer_size);
uint64_t COUNT4;
hipMemcpy(&COUNT4, d_COUNT4, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 1);
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_4, d_BUF_IDX_4, d_HT_4.ref(cuco::insert), d_customer__c_custkey, customer_size);
//Materialize count
uint64_t* d_COUNT6;
hipMalloc(&d_COUNT6, sizeof(uint64_t));
hipMemset(d_COUNT6, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)orders_size/128.), 128>>>(d_BUF_4, d_COUNT6, d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::find), d_orders__o_custkey, d_orders__o_orderkey, orders_size);
uint64_t COUNT6;
hipMemcpy(&COUNT6, d_COUNT6, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 2);
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)orders_size/128.), 128>>>(d_BUF_4, d_BUF_6, d_BUF_IDX_6, d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderkey, orders_size);
//Create aggregation hash table
auto d_HT_8 = cuco::static_map{ (int)6001215*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_9<<<std::ceil((float)lineitem_size/128.), 128>>>(d_BUF_6, d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::insert), d_lineitem__l_orderkey, lineitem_size, d_orders__o_orderkey);
size_t COUNT8 = d_HT_8.size();
thrust::device_vector<int64_t> keys_8(COUNT8), vals_8(COUNT8);
d_HT_8.retrieve_all(keys_8.begin(), vals_8.begin());
d_HT_8.clear();
int64_t* raw_keys8 = thrust::raw_pointer_cast(keys_8.data());
insertKeys<<<std::ceil((float)COUNT8/128.), 128>>>(raw_keys8, d_HT_8.ref(cuco::insert), COUNT8);
//Aggregate in hashtable
DBDecimalType* d_aggr1__tmp_attr1;
hipMalloc(&d_aggr1__tmp_attr1, sizeof(DBDecimalType) * COUNT8);
hipMemset(d_aggr1__tmp_attr1, 0, sizeof(DBDecimalType) * COUNT8);
DBI32Type* d_aggr__c_custkey;
hipMalloc(&d_aggr__c_custkey, sizeof(DBI32Type) * COUNT8);
hipMemset(d_aggr__c_custkey, 0, sizeof(DBI32Type) * COUNT8);
DBDecimalType* d_aggr__o_totalprice;
hipMalloc(&d_aggr__o_totalprice, sizeof(DBDecimalType) * COUNT8);
hipMemset(d_aggr__o_totalprice, 0, sizeof(DBDecimalType) * COUNT8);
DBDateType* d_aggr__o_orderdate;
hipMalloc(&d_aggr__o_orderdate, sizeof(DBDateType) * COUNT8);
hipMemset(d_aggr__o_orderdate, 0, sizeof(DBDateType) * COUNT8);
DBI32Type* d_KEY_8orders__o_orderkey;
hipMalloc(&d_KEY_8orders__o_orderkey, sizeof(DBI32Type) * COUNT8);
hipMemset(d_KEY_8orders__o_orderkey, 0, sizeof(DBI32Type) * COUNT8);
main_9<<<std::ceil((float)lineitem_size/128.), 128>>>(d_BUF_6, d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::find), d_KEY_8orders__o_orderkey, d_aggr1__tmp_attr1, d_aggr__c_custkey, d_aggr__o_orderdate, d_aggr__o_totalprice, d_customer__c_custkey, d_lineitem__l_orderkey, d_lineitem__l_quantity, lineitem_size, d_orders__o_orderdate, d_orders__o_orderkey, d_orders__o_totalprice);
//Materialize count
uint64_t* d_COUNT10;
hipMalloc(&d_COUNT10, sizeof(uint64_t));
hipMemset(d_COUNT10, 0, sizeof(uint64_t));
count_11<<<std::ceil((float)COUNT8/128.), 128>>>(d_COUNT10, COUNT8);
uint64_t COUNT10;
hipMemcpy(&COUNT10, d_COUNT10, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX10;
hipMalloc(&d_MAT_IDX10, sizeof(uint64_t));
hipMemset(d_MAT_IDX10, 0, sizeof(uint64_t));
auto MAT10aggr__c_custkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT10);
DBI32Type* d_MAT10aggr__c_custkey;
hipMalloc(&d_MAT10aggr__c_custkey, sizeof(DBI32Type) * COUNT10);
auto MAT10orders__o_orderkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT10);
DBI32Type* d_MAT10orders__o_orderkey;
hipMalloc(&d_MAT10orders__o_orderkey, sizeof(DBI32Type) * COUNT10);
auto MAT10aggr__o_orderdate = (DBDateType*)malloc(sizeof(DBDateType) * COUNT10);
DBDateType* d_MAT10aggr__o_orderdate;
hipMalloc(&d_MAT10aggr__o_orderdate, sizeof(DBDateType) * COUNT10);
auto MAT10aggr__o_totalprice = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT10);
DBDecimalType* d_MAT10aggr__o_totalprice;
hipMalloc(&d_MAT10aggr__o_totalprice, sizeof(DBDecimalType) * COUNT10);
auto MAT10aggr1__tmp_attr1 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT10);
DBDecimalType* d_MAT10aggr1__tmp_attr1;
hipMalloc(&d_MAT10aggr1__tmp_attr1, sizeof(DBDecimalType) * COUNT10);
main_11<<<std::ceil((float)COUNT8/128.), 128>>>(COUNT8, d_MAT10aggr1__tmp_attr1, d_MAT10aggr__c_custkey, d_MAT10aggr__o_orderdate, d_MAT10aggr__o_totalprice, d_MAT10orders__o_orderkey, d_MAT_IDX10, d_aggr1__tmp_attr1, d_aggr__c_custkey, d_aggr__o_orderdate, d_aggr__o_totalprice, d_KEY_8orders__o_orderkey);
hipMemcpy(MAT10aggr__c_custkey, d_MAT10aggr__c_custkey, sizeof(DBI32Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10orders__o_orderkey, d_MAT10orders__o_orderkey, sizeof(DBI32Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10aggr__o_orderdate, d_MAT10aggr__o_orderdate, sizeof(DBDateType) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10aggr__o_totalprice, d_MAT10aggr__o_totalprice, sizeof(DBDecimalType) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10aggr1__tmp_attr1, d_MAT10aggr1__tmp_attr1, sizeof(DBDecimalType) * COUNT10, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT10; i++) { std::cout << "" << MAT10aggr__c_custkey[i];
std::cout << "|" << MAT10orders__o_orderkey[i];
std::cout << "|" << MAT10aggr__o_orderdate[i];
std::cout << "|" << MAT10aggr__o_totalprice[i];
std::cout << "|" << MAT10aggr1__tmp_attr1[i];
std::cout << std::endl; }
hipFree(d_KEY_0lineitem_u_1__l_orderkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT2);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_COUNT4);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_COUNT6);
hipFree(d_KEY_8orders__o_orderkey);
hipFree(d_aggr1__tmp_attr1);
hipFree(d_aggr__c_custkey);
hipFree(d_aggr__o_orderdate);
hipFree(d_aggr__o_totalprice);
hipFree(d_COUNT10);
hipFree(d_MAT10aggr1__tmp_attr1);
hipFree(d_MAT10aggr__c_custkey);
hipFree(d_MAT10aggr__o_orderdate);
hipFree(d_MAT10aggr__o_totalprice);
hipFree(d_MAT10orders__o_orderkey);
hipFree(d_MAT_IDX10);
free(MAT10aggr1__tmp_attr1);
free(MAT10aggr__c_custkey);
free(MAT10aggr__o_orderdate);
free(MAT10aggr__o_totalprice);
free(MAT10orders__o_orderkey);
}