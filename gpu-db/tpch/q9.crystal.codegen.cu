#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_1(uint64_t* COUNT0, size_t nation_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT0, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBI32Type* nation__n_nationkey, size_t nation_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_nation__n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_nation__n_nationkey[ITEM] = nation__n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_nation__n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0[ITEM], buf_idx_0});
BUF_0[(buf_idx_0) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE>
__global__ void count_3(uint64_t* BUF_0, uint64_t* COUNT2, HASHTABLE_PROBE HT_0, DBI32Type* supplier__s_nationkey, size_t supplier_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_nationkey[ITEM] = supplier__s_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_supplier__s_nationkey[ITEM];
}
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT2, 1);
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_0, uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_PROBE HT_0, HASHTABLE_INSERT HT_2, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_nationkey[ITEM] = supplier__s_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_supplier__s_nationkey[ITEM];
}
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_2 = atomicAdd((int*)BUF_IDX_2, 1);
HT_2.insert(cuco::pair{KEY_2[ITEM], buf_idx_2});
BUF_2[(buf_idx_2) * 2 + 0] = ITEM*TB + tid;
BUF_2[(buf_idx_2) * 2 + 1] = BUF_0[slot_second0[ITEM] * 1 + 0];
}
}
__global__ void count_5(uint64_t* COUNT4, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT4, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_INSERT HT_4, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_orders__o_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_4 = atomicAdd((int*)BUF_IDX_4, 1);
HT_4.insert(cuco::pair{KEY_4[ITEM], buf_idx_4});
BUF_4[(buf_idx_4) * 1 + 0] = ITEM*TB + tid;
}
}
__global__ void count_7(uint64_t* COUNT6, size_t part_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT6, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_7(uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_INSERT HT_6, DBI32Type* part__p_partkey, size_t part_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_6 = atomicAdd((int*)BUF_IDX_6, 1);
HT_6.insert(cuco::pair{KEY_6[ITEM], buf_idx_6});
BUF_6[(buf_idx_6) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE>
__global__ void count_9(uint64_t* BUF_2, uint64_t* COUNT8, HASHTABLE_PROBE HT_2, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_partsupp__ps_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
reg_partsupp__ps_suppkey[ITEM] = partsupp__ps_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_partsupp__ps_suppkey[ITEM];
}
int64_t slot_second2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_2 = HT_2.find(KEY_2[ITEM]);
if (SLOT_2 == HT_2.end()) {selection_flags[ITEM] = 0; continue;}
slot_second2[ITEM] = SLOT_2->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT8, 1);
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_9(uint64_t* BUF_2, uint64_t* BUF_8, uint64_t* BUF_IDX_8, HASHTABLE_PROBE HT_2, HASHTABLE_INSERT HT_8, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size, DBI32Type* supplier__s_suppkey) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_partsupp__ps_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
reg_partsupp__ps_suppkey[ITEM] = partsupp__ps_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_partsupp__ps_suppkey[ITEM];
}
int64_t slot_second2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_2 = HT_2.find(KEY_2[ITEM]);
if (SLOT_2 == HT_2.end()) {selection_flags[ITEM] = 0; continue;}
slot_second2[ITEM] = SLOT_2->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[BUF_2[slot_second2[ITEM] * 2 + 0]];
}
DBI32Type reg_partsupp__ps_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
reg_partsupp__ps_partkey[ITEM] = partsupp__ps_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_supplier__s_suppkey[ITEM];
KEY_8[ITEM] <<= 32;
KEY_8[ITEM] |= reg_partsupp__ps_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_8 = atomicAdd((int*)BUF_IDX_8, 1);
HT_8.insert(cuco::pair{KEY_8[ITEM], buf_idx_8});
BUF_8[(buf_idx_8) * 3 + 0] = BUF_2[slot_second2[ITEM] * 2 + 0];
BUF_8[(buf_idx_8) * 3 + 1] = ITEM*TB + tid;
BUF_8[(buf_idx_8) * 3 + 2] = BUF_2[slot_second2[ITEM] * 2 + 1];
}
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_11(uint64_t* BUF_4, uint64_t* BUF_6, uint64_t* BUF_8, HASHTABLE_INSERT HT_10, HASHTABLE_PROBE HT_4, HASHTABLE_PROBE HT_6, HASHTABLE_PROBE HT_8, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded, DBDateType* orders__o_orderdate) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
int64_t slot_second4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0; continue;}
slot_second4[ITEM] = SLOT_4->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_suppkey[ITEM] = lineitem__l_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_lineitem__l_suppkey[ITEM];
KEY_8[ITEM] <<= 32;
KEY_8[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
int64_t slot_second8[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_8 = HT_8.find(KEY_8[ITEM]);
if (SLOT_8 == HT_8.end()) {selection_flags[ITEM] = 0; continue;}
slot_second8[ITEM] = SLOT_8->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI16Type reg_nation__n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_nation__n_name_encoded[ITEM] = nation__n_name_encoded[BUF_8[slot_second8[ITEM] * 3 + 2]];
}
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[BUF_4[slot_second4[ITEM] * 1 + 0]];
}
DBI64Type reg_map0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr0[ITEM] = ExtractFromDate("year", reg_orders__o_orderdate[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_nation__n_name_encoded[ITEM];
KEY_10[ITEM] <<= 32;
KEY_10[ITEM] |= (DBI32Type)reg_map0__tmp_attr0[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_10.insert(cuco::pair{KEY_10[ITEM], 1});
}
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_11(uint64_t* BUF_4, uint64_t* BUF_6, uint64_t* BUF_8, HASHTABLE_FIND HT_10, HASHTABLE_PROBE HT_4, HASHTABLE_PROBE HT_6, HASHTABLE_PROBE HT_8, DBI64Type* KEY_10map0__tmp_attr0, DBI16Type* KEY_10nation__n_name_encoded, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded, DBDateType* orders__o_orderdate, DBDecimalType* partsupp__ps_supplycost) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
int64_t slot_second4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0; continue;}
slot_second4[ITEM] = SLOT_4->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_suppkey[ITEM] = lineitem__l_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_lineitem__l_suppkey[ITEM];
KEY_8[ITEM] <<= 32;
KEY_8[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
int64_t slot_second8[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_8 = HT_8.find(KEY_8[ITEM]);
if (SLOT_8 == HT_8.end()) {selection_flags[ITEM] = 0; continue;}
slot_second8[ITEM] = SLOT_8->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI16Type reg_nation__n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_nation__n_name_encoded[ITEM] = nation__n_name_encoded[BUF_8[slot_second8[ITEM] * 3 + 2]];
}
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[BUF_4[slot_second4[ITEM] * 1 + 0]];
}
DBI64Type reg_map0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr0[ITEM] = ExtractFromDate("year", reg_orders__o_orderdate[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_nation__n_name_encoded[ITEM];
KEY_10[ITEM] <<= 32;
KEY_10[ITEM] |= (DBI32Type)reg_map0__tmp_attr0[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_quantity[ITEM] = lineitem__l_quantity[ITEM*TB + tid];
}
DBDecimalType reg_partsupp__ps_supplycost[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_partsupp__ps_supplycost[ITEM] = partsupp__ps_supplycost[BUF_8[slot_second8[ITEM] * 3 + 1]];
}
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = ((reg_lineitem__l_extendedprice[ITEM]) * ((1) - (reg_lineitem__l_discount[ITEM]))) - ((reg_partsupp__ps_supplycost[ITEM]) * (reg_lineitem__l_quantity[ITEM]));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_10 = HT_10.find(KEY_10[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr2[buf_idx_10], reg_map0__tmp_attr1[ITEM]);
KEY_10nation__n_name_encoded[buf_idx_10] = reg_nation__n_name_encoded[ITEM];
KEY_10map0__tmp_attr0[buf_idx_10] = reg_map0__tmp_attr0[ITEM];
}
}
__global__ void count_13(size_t COUNT10, uint64_t* COUNT12) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT12, 1);
}
}
__global__ void main_13(size_t COUNT10, DBDecimalType* MAT12aggr0__tmp_attr2, DBI64Type* MAT12map0__tmp_attr0, DBI16Type* MAT12nation__n_name_encoded, uint64_t* MAT_IDX12, DBDecimalType* aggr0__tmp_attr2, DBI64Type* map0__tmp_attr0, DBI16Type* nation__n_name_encoded) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize buffers
DBI16Type reg_nation__n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_nation__n_name_encoded[ITEM] = nation__n_name_encoded[ITEM*TB + tid];
}
DBI64Type reg_map0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_map0__tmp_attr0[ITEM] = map0__tmp_attr0[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_aggr0__tmp_attr2[ITEM] = aggr0__tmp_attr2[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx12 = atomicAdd((int*)MAT_IDX12, 1);
MAT12nation__n_name_encoded[mat_idx12] = reg_nation__n_name_encoded[ITEM];
MAT12map0__tmp_attr0[mat_idx12] = reg_map0__tmp_attr0[ITEM];
MAT12aggr0__tmp_attr2[mat_idx12] = reg_aggr0__tmp_attr2[ITEM];
}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map) {
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT0, nation_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_nation__n_nationkey, nation_size);
//Materialize count
uint64_t* d_COUNT2;
hipMalloc(&d_COUNT2, sizeof(uint64_t));
hipMemset(d_COUNT2, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_COUNT2, d_HT_0.ref(cuco::find), d_supplier__s_nationkey, supplier_size);
uint64_t COUNT2;
hipMemcpy(&COUNT2, d_COUNT2, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 2);
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_BUF_2, d_BUF_IDX_2, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT4;
hipMalloc(&d_COUNT4, sizeof(uint64_t));
hipMemset(d_COUNT4, 0, sizeof(uint64_t));
count_5<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT4, orders_size);
uint64_t COUNT4;
hipMemcpy(&COUNT4, d_COUNT4, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 1);
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_BUF_IDX_4, d_HT_4.ref(cuco::insert), d_orders__o_orderkey, orders_size);
//Materialize count
uint64_t* d_COUNT6;
hipMalloc(&d_COUNT6, sizeof(uint64_t));
hipMemset(d_COUNT6, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)part_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT6, part_size);
uint64_t COUNT6;
hipMemcpy(&COUNT6, d_COUNT6, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 1);
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)part_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_6, d_BUF_IDX_6, d_HT_6.ref(cuco::insert), d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT8;
hipMalloc(&d_COUNT8, sizeof(uint64_t));
hipMemset(d_COUNT8, 0, sizeof(uint64_t));
count_9<<<std::ceil((float)partsupp_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_2, d_COUNT8, d_HT_2.ref(cuco::find), d_partsupp__ps_suppkey, partsupp_size);
uint64_t COUNT8;
hipMemcpy(&COUNT8, d_COUNT8, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_8;
hipMalloc(&d_BUF_IDX_8, sizeof(uint64_t));
hipMemset(d_BUF_IDX_8, 0, sizeof(uint64_t));
uint64_t* d_BUF_8;
hipMalloc(&d_BUF_8, sizeof(uint64_t) * COUNT8 * 3);
auto d_HT_8 = cuco::static_map{ (int)COUNT8*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_9<<<std::ceil((float)partsupp_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_2, d_BUF_8, d_BUF_IDX_8, d_HT_2.ref(cuco::find), d_HT_8.ref(cuco::insert), d_partsupp__ps_partkey, d_partsupp__ps_suppkey, partsupp_size, d_supplier__s_suppkey);
//Create aggregation hash table
auto d_HT_10 = cuco::static_map{ (int)48009721*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_11<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_BUF_6, d_BUF_8, d_HT_10.ref(cuco::insert), d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::find), d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded, d_orders__o_orderdate);
size_t COUNT10 = d_HT_10.size();
thrust::device_vector<int64_t> keys_10(COUNT10), vals_10(COUNT10);
d_HT_10.retrieve_all(keys_10.begin(), vals_10.begin());
d_HT_10.clear();
int64_t* raw_keys10 = thrust::raw_pointer_cast(keys_10.data());
insertKeys<<<std::ceil((float)COUNT10/128.), 128>>>(raw_keys10, d_HT_10.ref(cuco::insert), COUNT10);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT10);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT10);
DBI16Type* d_KEY_10nation__n_name_encoded;
hipMalloc(&d_KEY_10nation__n_name_encoded, sizeof(DBI16Type) * COUNT10);
hipMemset(d_KEY_10nation__n_name_encoded, 0, sizeof(DBI16Type) * COUNT10);
DBI64Type* d_KEY_10map0__tmp_attr0;
hipMalloc(&d_KEY_10map0__tmp_attr0, sizeof(DBI64Type) * COUNT10);
hipMemset(d_KEY_10map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT10);
main_11<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_BUF_6, d_BUF_8, d_HT_10.ref(cuco::find), d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::find), d_KEY_10map0__tmp_attr0, d_KEY_10nation__n_name_encoded, d_aggr0__tmp_attr2, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_quantity, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded, d_orders__o_orderdate, d_partsupp__ps_supplycost);
//Materialize count
uint64_t* d_COUNT12;
hipMalloc(&d_COUNT12, sizeof(uint64_t));
hipMemset(d_COUNT12, 0, sizeof(uint64_t));
count_13<<<std::ceil((float)COUNT10/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT10, d_COUNT12);
uint64_t COUNT12;
hipMemcpy(&COUNT12, d_COUNT12, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX12;
hipMalloc(&d_MAT_IDX12, sizeof(uint64_t));
hipMemset(d_MAT_IDX12, 0, sizeof(uint64_t));
auto MAT12nation__n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT12);
DBI16Type* d_MAT12nation__n_name_encoded;
hipMalloc(&d_MAT12nation__n_name_encoded, sizeof(DBI16Type) * COUNT12);
auto MAT12map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT12);
DBI64Type* d_MAT12map0__tmp_attr0;
hipMalloc(&d_MAT12map0__tmp_attr0, sizeof(DBI64Type) * COUNT12);
auto MAT12aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT12);
DBDecimalType* d_MAT12aggr0__tmp_attr2;
hipMalloc(&d_MAT12aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT12);
main_13<<<std::ceil((float)COUNT10/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT10, d_MAT12aggr0__tmp_attr2, d_MAT12map0__tmp_attr0, d_MAT12nation__n_name_encoded, d_MAT_IDX12, d_aggr0__tmp_attr2, d_KEY_10map0__tmp_attr0, d_KEY_10nation__n_name_encoded);
hipMemcpy(MAT12nation__n_name_encoded, d_MAT12nation__n_name_encoded, sizeof(DBI16Type) * COUNT12, hipMemcpyDeviceToHost);
hipMemcpy(MAT12map0__tmp_attr0, d_MAT12map0__tmp_attr0, sizeof(DBI64Type) * COUNT12, hipMemcpyDeviceToHost);
hipMemcpy(MAT12aggr0__tmp_attr2, d_MAT12aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT12, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT12; i++) { std::cout << "" << nation__n_name_map[MAT12nation__n_name_encoded[i]];
std::cout << "," << MAT12map0__tmp_attr0[i];
std::cout << "," << MAT12aggr0__tmp_attr2[i];
std::cout << std::endl; }
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_COUNT0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_COUNT2);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_COUNT4);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_COUNT6);
hipFree(d_BUF_8);
hipFree(d_BUF_IDX_8);
hipFree(d_COUNT8);
hipFree(d_KEY_10map0__tmp_attr0);
hipFree(d_KEY_10nation__n_name_encoded);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_COUNT12);
hipFree(d_MAT12aggr0__tmp_attr2);
hipFree(d_MAT12map0__tmp_attr0);
hipFree(d_MAT12nation__n_name_encoded);
hipFree(d_MAT_IDX12);
free(MAT12aggr0__tmp_attr2);
free(MAT12map0__tmp_attr0);
free(MAT12nation__n_name_encoded);
}