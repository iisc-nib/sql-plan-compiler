#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_FIND>
__global__ void main_1(HASHTABLE_FIND HT_0, DBI32Type* KEY_0lineitem_u_1__l_orderkey, int* SLOT_COUNT_0, DBDecimalType* aggr0__tmp_attr0, size_t lineitem_size, DBI32Type* lineitem_u_1__l_orderkey, DBDecimalType* lineitem_u_1__l_quantity) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_lineitem_u_1__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem_u_1__l_orderkey[ITEM] = lineitem_u_1__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_lineitem_u_1__l_orderkey[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem_u_1__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem_u_1__l_quantity[ITEM] = lineitem_u_1__l_quantity[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_0 = get_aggregation_slot(KEY_0[ITEM], HT_0, SLOT_COUNT_0);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_0], reg_lineitem_u_1__l_quantity[ITEM]);
KEY_0lineitem_u_1__l_orderkey[buf_idx_0] = reg_lineitem_u_1__l_orderkey[ITEM];
}
}
template<typename HASHTABLE_INSERT_SJ>
__global__ void main_3(size_t COUNT0, HASHTABLE_INSERT_SJ HT_2, DBDecimalType* aggr0__tmp_attr0, DBI32Type* lineitem_u_1__l_orderkey) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_aggr0__tmp_attr0[ITEM], 300.0, Predicate::gt);
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_lineitem_u_1__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
reg_lineitem_u_1__l_orderkey[ITEM] = lineitem_u_1__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_lineitem_u_1__l_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT0); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_2.insert(cuco::pair{KEY_2[ITEM], 1});
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_INSERT HT_4, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_4.insert(cuco::pair{KEY_4[ITEM], ITEM*TB + tid});
BUF_4[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE_SJ, typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_7(uint64_t* BUF_4, uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_PROBE_SJ HT_2, HASHTABLE_PROBE HT_4, HASHTABLE_INSERT HT_6, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_orders__o_orderkey[ITEM];
}
//Probe Hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_2 = HT_2.find(KEY_2[ITEM]);
if (SLOT_2 == HT_2.end()) {selection_flags[ITEM] = 0;}
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_custkey[ITEM] = orders__o_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_orders__o_custkey[ITEM];
}
//Probe Hash table
int64_t slot_second4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0; continue;}
slot_second4[ITEM] = SLOT_4->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_orders__o_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_6.insert(cuco::pair{KEY_6[ITEM], ITEM*TB + tid});
BUF_6[(ITEM*TB + tid) * 2 + 0] = BUF_4[slot_second4[ITEM] * 1 + 0];
BUF_6[(ITEM*TB + tid) * 2 + 1] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_9(uint64_t* BUF_6, HASHTABLE_PROBE HT_6, HASHTABLE_FIND HT_8, DBI32Type* KEY_8orders__o_orderkey, int* SLOT_COUNT_8, DBDecimalType* aggr1__tmp_attr1, DBI32Type* aggr__c_custkey, DBDateType* aggr__o_orderdate, DBDecimalType* aggr__o_totalprice, DBI32Type* customer__c_custkey, DBI32Type* lineitem__l_orderkey, DBDecimalType* lineitem__l_quantity, size_t lineitem_size, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, DBDecimalType* orders__o_totalprice) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
//Probe Hash table
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[BUF_6[slot_second6[ITEM] * 2 + 1]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_orders__o_orderkey[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_quantity[ITEM] = lineitem__l_quantity[ITEM*TB + tid];
}
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_custkey[ITEM] = customer__c_custkey[BUF_6[slot_second6[ITEM] * 2 + 0]];
}
DBDecimalType reg_orders__o_totalprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_totalprice[ITEM] = orders__o_totalprice[BUF_6[slot_second6[ITEM] * 2 + 1]];
}
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[BUF_6[slot_second6[ITEM] * 2 + 1]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_8 = get_aggregation_slot(KEY_8[ITEM], HT_8, SLOT_COUNT_8);
aggregate_sum(&aggr1__tmp_attr1[buf_idx_8], reg_lineitem__l_quantity[ITEM]);
aggregate_any(&aggr__c_custkey[buf_idx_8], reg_customer__c_custkey[ITEM]);
aggregate_any(&aggr__o_totalprice[buf_idx_8], reg_orders__o_totalprice[ITEM]);
aggregate_any(&aggr__o_orderdate[buf_idx_8], reg_orders__o_orderdate[ITEM]);
KEY_8orders__o_orderkey[buf_idx_8] = reg_orders__o_orderkey[ITEM];
}
}
__global__ void main_11(size_t COUNT8, DBDecimalType* MAT10aggr1__tmp_attr1, DBI32Type* MAT10aggr__c_custkey, DBDateType* MAT10aggr__o_orderdate, DBDecimalType* MAT10aggr__o_totalprice, DBI32Type* MAT10orders__o_orderkey, uint64_t* MAT_IDX10, DBDecimalType* aggr1__tmp_attr1, DBI32Type* aggr__c_custkey, DBDateType* aggr__o_orderdate, DBDecimalType* aggr__o_totalprice, DBI32Type* orders__o_orderkey) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize buffers
DBI32Type reg_aggr__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT8); ++ITEM) {
reg_aggr__c_custkey[ITEM] = aggr__c_custkey[ITEM*TB + tid];
}
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT8); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
DBDateType reg_aggr__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT8); ++ITEM) {
reg_aggr__o_orderdate[ITEM] = aggr__o_orderdate[ITEM*TB + tid];
}
DBDecimalType reg_aggr__o_totalprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT8); ++ITEM) {
reg_aggr__o_totalprice[ITEM] = aggr__o_totalprice[ITEM*TB + tid];
}
DBDecimalType reg_aggr1__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT8); ++ITEM) {
reg_aggr1__tmp_attr1[ITEM] = aggr1__tmp_attr1[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT8); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx10 = atomicAdd((int*)MAT_IDX10, 1);
MAT10aggr__c_custkey[mat_idx10] = reg_aggr__c_custkey[ITEM];
MAT10orders__o_orderkey[mat_idx10] = reg_orders__o_orderkey[ITEM];
MAT10aggr__o_orderdate[mat_idx10] = reg_aggr__o_orderdate[ITEM];
MAT10aggr__o_totalprice[mat_idx10] = reg_aggr__o_totalprice[ITEM];
MAT10aggr1__tmp_attr1[mat_idx10] = reg_aggr1__tmp_attr1[ITEM];
}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t COUNT0 = 6001215;
auto d_HT_0 = cuco::static_map{ (int)6001215*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_0;
hipMalloc(&d_SLOT_COUNT_0, sizeof(int));
hipMemset(d_SLOT_COUNT_0, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT0);
DBI32Type* d_KEY_0lineitem_u_1__l_orderkey;
hipMalloc(&d_KEY_0lineitem_u_1__l_orderkey, sizeof(DBI32Type) * COUNT0);
hipMemset(d_KEY_0lineitem_u_1__l_orderkey, 0, sizeof(DBI32Type) * COUNT0);
main_1<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_HT_0.ref(cuco::insert_and_find), d_KEY_0lineitem_u_1__l_orderkey, d_SLOT_COUNT_0, d_aggr0__tmp_attr0, lineitem_size, d_lineitem__l_orderkey, d_lineitem__l_quantity);
COUNT0 = d_HT_0.size();
size_t COUNT2 = COUNT0;
// Insert hash table control;
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)COUNT0/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT0, d_HT_2.ref(cuco::insert), d_aggr0__tmp_attr0, d_KEY_0lineitem_u_1__l_orderkey);
size_t COUNT4 = customer_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 1);
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_BUF_IDX_4, d_HT_4.ref(cuco::insert), d_customer__c_custkey, customer_size);
size_t COUNT6 = orders_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 2);
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_BUF_6, d_BUF_IDX_6, d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderkey, orders_size);
size_t COUNT8 = 6001215;
auto d_HT_8 = cuco::static_map{ (int)6001215*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_8;
hipMalloc(&d_SLOT_COUNT_8, sizeof(int));
hipMemset(d_SLOT_COUNT_8, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr1__tmp_attr1;
hipMalloc(&d_aggr1__tmp_attr1, sizeof(DBDecimalType) * COUNT8);
hipMemset(d_aggr1__tmp_attr1, 0, sizeof(DBDecimalType) * COUNT8);
DBI32Type* d_aggr__c_custkey;
hipMalloc(&d_aggr__c_custkey, sizeof(DBI32Type) * COUNT8);
hipMemset(d_aggr__c_custkey, 0, sizeof(DBI32Type) * COUNT8);
DBDecimalType* d_aggr__o_totalprice;
hipMalloc(&d_aggr__o_totalprice, sizeof(DBDecimalType) * COUNT8);
hipMemset(d_aggr__o_totalprice, 0, sizeof(DBDecimalType) * COUNT8);
DBDateType* d_aggr__o_orderdate;
hipMalloc(&d_aggr__o_orderdate, sizeof(DBDateType) * COUNT8);
hipMemset(d_aggr__o_orderdate, 0, sizeof(DBDateType) * COUNT8);
DBI32Type* d_KEY_8orders__o_orderkey;
hipMalloc(&d_KEY_8orders__o_orderkey, sizeof(DBI32Type) * COUNT8);
hipMemset(d_KEY_8orders__o_orderkey, 0, sizeof(DBI32Type) * COUNT8);
main_9<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_6, d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::insert_and_find), d_KEY_8orders__o_orderkey, d_SLOT_COUNT_8, d_aggr1__tmp_attr1, d_aggr__c_custkey, d_aggr__o_orderdate, d_aggr__o_totalprice, d_customer__c_custkey, d_lineitem__l_orderkey, d_lineitem__l_quantity, lineitem_size, d_orders__o_orderdate, d_orders__o_orderkey, d_orders__o_totalprice);
COUNT8 = d_HT_8.size();
size_t COUNT10 = COUNT8;
//Materialize buffers
uint64_t* d_MAT_IDX10;
hipMalloc(&d_MAT_IDX10, sizeof(uint64_t));
hipMemset(d_MAT_IDX10, 0, sizeof(uint64_t));
auto MAT10aggr__c_custkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT10);
DBI32Type* d_MAT10aggr__c_custkey;
hipMalloc(&d_MAT10aggr__c_custkey, sizeof(DBI32Type) * COUNT10);
auto MAT10orders__o_orderkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT10);
DBI32Type* d_MAT10orders__o_orderkey;
hipMalloc(&d_MAT10orders__o_orderkey, sizeof(DBI32Type) * COUNT10);
auto MAT10aggr__o_orderdate = (DBDateType*)malloc(sizeof(DBDateType) * COUNT10);
DBDateType* d_MAT10aggr__o_orderdate;
hipMalloc(&d_MAT10aggr__o_orderdate, sizeof(DBDateType) * COUNT10);
auto MAT10aggr__o_totalprice = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT10);
DBDecimalType* d_MAT10aggr__o_totalprice;
hipMalloc(&d_MAT10aggr__o_totalprice, sizeof(DBDecimalType) * COUNT10);
auto MAT10aggr1__tmp_attr1 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT10);
DBDecimalType* d_MAT10aggr1__tmp_attr1;
hipMalloc(&d_MAT10aggr1__tmp_attr1, sizeof(DBDecimalType) * COUNT10);
main_11<<<std::ceil((float)COUNT8/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT8, d_MAT10aggr1__tmp_attr1, d_MAT10aggr__c_custkey, d_MAT10aggr__o_orderdate, d_MAT10aggr__o_totalprice, d_MAT10orders__o_orderkey, d_MAT_IDX10, d_aggr1__tmp_attr1, d_aggr__c_custkey, d_aggr__o_orderdate, d_aggr__o_totalprice, d_KEY_8orders__o_orderkey);
uint64_t MATCOUNT_10 = 0;
hipMemcpy(&MATCOUNT_10, d_MAT_IDX10, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT10aggr__c_custkey, d_MAT10aggr__c_custkey, sizeof(DBI32Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10orders__o_orderkey, d_MAT10orders__o_orderkey, sizeof(DBI32Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10aggr__o_orderdate, d_MAT10aggr__o_orderdate, sizeof(DBDateType) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10aggr__o_totalprice, d_MAT10aggr__o_totalprice, sizeof(DBDecimalType) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10aggr1__tmp_attr1, d_MAT10aggr1__tmp_attr1, sizeof(DBDecimalType) * COUNT10, hipMemcpyDeviceToHost);
for (auto i=0ull; i < MATCOUNT_10; i++) { std::cout << "" << MAT10aggr__c_custkey[i];
std::cout << "|" << MAT10orders__o_orderkey[i];
std::cout << "|" << MAT10aggr__o_orderdate[i];
std::cout << "|" << MAT10aggr__o_totalprice[i];
std::cout << "|" << MAT10aggr1__tmp_attr1[i];
std::cout << std::endl; }
hipFree(d_KEY_0lineitem_u_1__l_orderkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_KEY_8orders__o_orderkey);
hipFree(d_aggr1__tmp_attr1);
hipFree(d_aggr__c_custkey);
hipFree(d_aggr__o_orderdate);
hipFree(d_aggr__o_totalprice);
hipFree(d_MAT10aggr1__tmp_attr1);
hipFree(d_MAT10aggr__c_custkey);
hipFree(d_MAT10aggr__o_orderdate);
hipFree(d_MAT10aggr__o_totalprice);
hipFree(d_MAT10orders__o_orderkey);
hipFree(d_MAT_IDX10);
free(MAT10aggr1__tmp_attr1);
free(MAT10aggr__c_custkey);
free(MAT10aggr__o_orderdate);
free(MAT10aggr__o_totalprice);
free(MAT10orders__o_orderkey);
}