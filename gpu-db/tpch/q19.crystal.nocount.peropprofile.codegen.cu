#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_INSERT>
__global__ void main_7(uint64_t* BUF_14, uint64_t* BUF_IDX_14, HASHTABLE_INSERT HT_14, int64_t* cycles_per_warp_main_7_join_build_14, int64_t* cycles_per_warp_main_7_selection_10, int64_t* cycles_per_warp_main_7_selection_11, int64_t* cycles_per_warp_main_7_selection_12, int64_t* cycles_per_warp_main_7_selection_13, int64_t* cycles_per_warp_main_7_selection_6, int64_t* cycles_per_warp_main_7_selection_8, int64_t* cycles_per_warp_main_7_selection_9, DBStringType* part__p_brand, DBStringType* part__p_container, DBI32Type* part__p_partkey, DBI32Type* part__p_size, size_t part_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_part__p_brand[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_brand[ITEM] = part__p_brand[ITEM*TB + tid];
}
DBI32Type reg_part__p_size[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_size[ITEM] = part__p_size[ITEM*TB + tid];
}
DBStringType reg_part__p_container[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_container[ITEM] = part__p_container[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_selection_6[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_selection_8[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_selection_9[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_selection_10[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_selection_11[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_selection_12[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_selection_13[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_14[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_14[ITEM] = 0;
KEY_14[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_14.insert(cuco::pair{KEY_14[ITEM], ITEM*TB + tid});
BUF_14[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_build_14[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_1(uint64_t* BUF_14, HASHTABLE_PROBE HT_14, HASHTABLE_FIND HT_16, int* SLOT_COUNT_16, DBDecimalType* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_1_aggregation_16, int64_t* cycles_per_warp_main_1_join_probe_14, int64_t* cycles_per_warp_main_1_map_15, int64_t* cycles_per_warp_main_1_selection_0, int64_t* cycles_per_warp_main_1_selection_2, int64_t* cycles_per_warp_main_1_selection_3, int64_t* cycles_per_warp_main_1_selection_4, int64_t* cycles_per_warp_main_1_selection_5, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, DBStringType* lineitem__l_shipinstruct, DBStringType* lineitem__l_shipmode, size_t lineitem_size, DBStringType* part__p_brand, DBStringType* part__p_container, DBI32Type* part__p_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_lineitem__l_shipinstruct[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipinstruct[ITEM] = lineitem__l_shipinstruct[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_shipinstruct[ITEM], "DELIVER IN PERSON", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
DBDecimalType reg_lineitem__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_quantity[ITEM] = lineitem__l_quantity[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 1.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 11.00, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 10.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 30.00, Predicate::lte));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 1.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 11.00, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 10.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 30.00, Predicate::lte));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_3[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 1.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 11.00, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 10.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 30.00, Predicate::lte));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
DBStringType reg_lineitem__l_shipmode[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipmode[ITEM] = lineitem__l_shipmode[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_lineitem__l_shipmode[ITEM], "AIR", Predicate::eq)) || (evaluatePredicate(reg_lineitem__l_shipmode[ITEM], "AIR REG", Predicate::eq));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_5[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_14[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_14[ITEM] = 0;
KEY_14[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
//Probe Hash table
int64_t slot_second14[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_14 = HT_14.find(KEY_14[ITEM]);
if (SLOT_14 == HT_14.end()) {selection_flags[ITEM] = 0; continue;}
slot_second14[ITEM] = SLOT_14->second;
}
DBStringType reg_part__p_brand[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_brand[ITEM] = part__p_brand[BUF_14[slot_second14[ITEM] * 1 + 0]];
}
DBI32Type reg_part__p_size[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_size[ITEM] = part__p_size[BUF_14[slot_second14[ITEM] * 1 + 0]];
}
DBStringType reg_part__p_container[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_container[ITEM] = part__p_container[BUF_14[slot_second14[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (((evaluatePredicate(reg_lineitem__l_quantity[ITEM], 1.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 11.00, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_lineitem__l_quantity[ITEM], 10.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq)))) || ((evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 30.00, Predicate::lte)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)))) && (true);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_probe_14[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_map_15[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_16[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_16[ITEM] = 0;
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_16 = get_aggregation_slot(KEY_16[ITEM], HT_16, SLOT_COUNT_16);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_16], reg_map0__tmp_attr1[ITEM]);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_aggregation_16[blockIdx.x] = cycles_per_warp;}
}
__global__ void main_18(size_t COUNT16, DBDecimalType* MAT17aggr0__tmp_attr0, uint64_t* MAT_IDX17, DBDecimalType* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_18_materialize_17) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT16); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT16); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx17 = atomicAdd((int*)MAT_IDX17, 1);
MAT17aggr0__tmp_attr0[mat_idx17] = reg_aggr0__tmp_attr0[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_18_materialize_17[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_7_selection_6;
auto main_7_selection_6_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_selection_6, sizeof(int64_t) * main_7_selection_6_cpw_size);
hipMemset(d_cycles_per_warp_main_7_selection_6, -1, sizeof(int64_t) * main_7_selection_6_cpw_size);
int64_t* d_cycles_per_warp_main_7_selection_8;
auto main_7_selection_8_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_selection_8, sizeof(int64_t) * main_7_selection_8_cpw_size);
hipMemset(d_cycles_per_warp_main_7_selection_8, -1, sizeof(int64_t) * main_7_selection_8_cpw_size);
int64_t* d_cycles_per_warp_main_7_selection_9;
auto main_7_selection_9_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_selection_9, sizeof(int64_t) * main_7_selection_9_cpw_size);
hipMemset(d_cycles_per_warp_main_7_selection_9, -1, sizeof(int64_t) * main_7_selection_9_cpw_size);
int64_t* d_cycles_per_warp_main_7_selection_10;
auto main_7_selection_10_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_selection_10, sizeof(int64_t) * main_7_selection_10_cpw_size);
hipMemset(d_cycles_per_warp_main_7_selection_10, -1, sizeof(int64_t) * main_7_selection_10_cpw_size);
int64_t* d_cycles_per_warp_main_7_selection_11;
auto main_7_selection_11_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_selection_11, sizeof(int64_t) * main_7_selection_11_cpw_size);
hipMemset(d_cycles_per_warp_main_7_selection_11, -1, sizeof(int64_t) * main_7_selection_11_cpw_size);
int64_t* d_cycles_per_warp_main_7_selection_12;
auto main_7_selection_12_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_selection_12, sizeof(int64_t) * main_7_selection_12_cpw_size);
hipMemset(d_cycles_per_warp_main_7_selection_12, -1, sizeof(int64_t) * main_7_selection_12_cpw_size);
int64_t* d_cycles_per_warp_main_7_selection_13;
auto main_7_selection_13_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_selection_13, sizeof(int64_t) * main_7_selection_13_cpw_size);
hipMemset(d_cycles_per_warp_main_7_selection_13, -1, sizeof(int64_t) * main_7_selection_13_cpw_size);
int64_t* d_cycles_per_warp_main_7_join_build_14;
auto main_7_join_build_14_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_build_14, sizeof(int64_t) * main_7_join_build_14_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_build_14, -1, sizeof(int64_t) * main_7_join_build_14_cpw_size);
size_t COUNT14 = part_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_14;
hipMalloc(&d_BUF_IDX_14, sizeof(uint64_t));
hipMemset(d_BUF_IDX_14, 0, sizeof(uint64_t));
uint64_t* d_BUF_14;
hipMalloc(&d_BUF_14, sizeof(uint64_t) * COUNT14 * 1);
auto d_HT_14 = cuco::static_map{ (int)COUNT14*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)part_size/(float)TILE_SIZE), TB>>>(d_BUF_14, d_BUF_IDX_14, d_HT_14.ref(cuco::insert), d_cycles_per_warp_main_7_join_build_14, d_cycles_per_warp_main_7_selection_10, d_cycles_per_warp_main_7_selection_11, d_cycles_per_warp_main_7_selection_12, d_cycles_per_warp_main_7_selection_13, d_cycles_per_warp_main_7_selection_6, d_cycles_per_warp_main_7_selection_8, d_cycles_per_warp_main_7_selection_9, d_part__p_brand, d_part__p_container, d_part__p_partkey, d_part__p_size, part_size);
int64_t* cycles_per_warp_main_7_selection_6 = (int64_t*)malloc(sizeof(int64_t) * main_7_selection_6_cpw_size);
hipMemcpy(cycles_per_warp_main_7_selection_6, d_cycles_per_warp_main_7_selection_6, sizeof(int64_t) * main_7_selection_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_selection_6 ";
for (auto i=0ull; i < main_7_selection_6_cpw_size; i++) std::cout << cycles_per_warp_main_7_selection_6[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_selection_8 = (int64_t*)malloc(sizeof(int64_t) * main_7_selection_8_cpw_size);
hipMemcpy(cycles_per_warp_main_7_selection_8, d_cycles_per_warp_main_7_selection_8, sizeof(int64_t) * main_7_selection_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_selection_8 ";
for (auto i=0ull; i < main_7_selection_8_cpw_size; i++) std::cout << cycles_per_warp_main_7_selection_8[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_selection_9 = (int64_t*)malloc(sizeof(int64_t) * main_7_selection_9_cpw_size);
hipMemcpy(cycles_per_warp_main_7_selection_9, d_cycles_per_warp_main_7_selection_9, sizeof(int64_t) * main_7_selection_9_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_selection_9 ";
for (auto i=0ull; i < main_7_selection_9_cpw_size; i++) std::cout << cycles_per_warp_main_7_selection_9[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_selection_10 = (int64_t*)malloc(sizeof(int64_t) * main_7_selection_10_cpw_size);
hipMemcpy(cycles_per_warp_main_7_selection_10, d_cycles_per_warp_main_7_selection_10, sizeof(int64_t) * main_7_selection_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_selection_10 ";
for (auto i=0ull; i < main_7_selection_10_cpw_size; i++) std::cout << cycles_per_warp_main_7_selection_10[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_selection_11 = (int64_t*)malloc(sizeof(int64_t) * main_7_selection_11_cpw_size);
hipMemcpy(cycles_per_warp_main_7_selection_11, d_cycles_per_warp_main_7_selection_11, sizeof(int64_t) * main_7_selection_11_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_selection_11 ";
for (auto i=0ull; i < main_7_selection_11_cpw_size; i++) std::cout << cycles_per_warp_main_7_selection_11[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_selection_12 = (int64_t*)malloc(sizeof(int64_t) * main_7_selection_12_cpw_size);
hipMemcpy(cycles_per_warp_main_7_selection_12, d_cycles_per_warp_main_7_selection_12, sizeof(int64_t) * main_7_selection_12_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_selection_12 ";
for (auto i=0ull; i < main_7_selection_12_cpw_size; i++) std::cout << cycles_per_warp_main_7_selection_12[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_selection_13 = (int64_t*)malloc(sizeof(int64_t) * main_7_selection_13_cpw_size);
hipMemcpy(cycles_per_warp_main_7_selection_13, d_cycles_per_warp_main_7_selection_13, sizeof(int64_t) * main_7_selection_13_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_selection_13 ";
for (auto i=0ull; i < main_7_selection_13_cpw_size; i++) std::cout << cycles_per_warp_main_7_selection_13[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_join_build_14 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_build_14_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_build_14, d_cycles_per_warp_main_7_join_build_14, sizeof(int64_t) * main_7_join_build_14_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_build_14 ";
for (auto i=0ull; i < main_7_join_build_14_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_build_14[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_2;
auto main_1_selection_2_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_2, -1, sizeof(int64_t) * main_1_selection_2_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_3;
auto main_1_selection_3_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_3, -1, sizeof(int64_t) * main_1_selection_3_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_4;
auto main_1_selection_4_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_4, sizeof(int64_t) * main_1_selection_4_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_4, -1, sizeof(int64_t) * main_1_selection_4_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_5;
auto main_1_selection_5_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_5, sizeof(int64_t) * main_1_selection_5_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_5, -1, sizeof(int64_t) * main_1_selection_5_cpw_size);
int64_t* d_cycles_per_warp_main_1_join_probe_14;
auto main_1_join_probe_14_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_probe_14, sizeof(int64_t) * main_1_join_probe_14_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_probe_14, -1, sizeof(int64_t) * main_1_join_probe_14_cpw_size);
int64_t* d_cycles_per_warp_main_1_map_15;
auto main_1_map_15_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_map_15, sizeof(int64_t) * main_1_map_15_cpw_size);
hipMemset(d_cycles_per_warp_main_1_map_15, -1, sizeof(int64_t) * main_1_map_15_cpw_size);
int64_t* d_cycles_per_warp_main_1_aggregation_16;
auto main_1_aggregation_16_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_aggregation_16, sizeof(int64_t) * main_1_aggregation_16_cpw_size);
hipMemset(d_cycles_per_warp_main_1_aggregation_16, -1, sizeof(int64_t) * main_1_aggregation_16_cpw_size);
size_t COUNT16 = 1;
auto d_HT_16 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_16;
hipMalloc(&d_SLOT_COUNT_16, sizeof(int));
hipMemset(d_SLOT_COUNT_16, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT16);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT16);
main_1<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TB>>>(d_BUF_14, d_HT_14.ref(cuco::find), d_HT_16.ref(cuco::insert_and_find), d_SLOT_COUNT_16, d_aggr0__tmp_attr0, d_cycles_per_warp_main_1_aggregation_16, d_cycles_per_warp_main_1_join_probe_14, d_cycles_per_warp_main_1_map_15, d_cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_3, d_cycles_per_warp_main_1_selection_4, d_cycles_per_warp_main_1_selection_5, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_partkey, d_lineitem__l_quantity, d_lineitem__l_shipinstruct, d_lineitem__l_shipmode, lineitem_size, d_part__p_brand, d_part__p_container, d_part__p_size);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_2 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_2 ";
for (auto i=0ull; i < main_1_selection_2_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_3 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_3, d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_3 ";
for (auto i=0ull; i < main_1_selection_3_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_3[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_4 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_4_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_4, d_cycles_per_warp_main_1_selection_4, sizeof(int64_t) * main_1_selection_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_4 ";
for (auto i=0ull; i < main_1_selection_4_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_5 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_5_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_5, d_cycles_per_warp_main_1_selection_5, sizeof(int64_t) * main_1_selection_5_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_5 ";
for (auto i=0ull; i < main_1_selection_5_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_5[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_probe_14 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_probe_14_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_probe_14, d_cycles_per_warp_main_1_join_probe_14, sizeof(int64_t) * main_1_join_probe_14_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_probe_14 ";
for (auto i=0ull; i < main_1_join_probe_14_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_probe_14[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_map_15 = (int64_t*)malloc(sizeof(int64_t) * main_1_map_15_cpw_size);
hipMemcpy(cycles_per_warp_main_1_map_15, d_cycles_per_warp_main_1_map_15, sizeof(int64_t) * main_1_map_15_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_map_15 ";
for (auto i=0ull; i < main_1_map_15_cpw_size; i++) std::cout << cycles_per_warp_main_1_map_15[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_aggregation_16 = (int64_t*)malloc(sizeof(int64_t) * main_1_aggregation_16_cpw_size);
hipMemcpy(cycles_per_warp_main_1_aggregation_16, d_cycles_per_warp_main_1_aggregation_16, sizeof(int64_t) * main_1_aggregation_16_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_aggregation_16 ";
for (auto i=0ull; i < main_1_aggregation_16_cpw_size; i++) std::cout << cycles_per_warp_main_1_aggregation_16[i] << " ";
std::cout << std::endl;
COUNT16 = d_HT_16.size();
int64_t* d_cycles_per_warp_main_18_materialize_17;
auto main_18_materialize_17_cpw_size = std::ceil((float)COUNT16/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_18_materialize_17, sizeof(int64_t) * main_18_materialize_17_cpw_size);
hipMemset(d_cycles_per_warp_main_18_materialize_17, -1, sizeof(int64_t) * main_18_materialize_17_cpw_size);
size_t COUNT17 = COUNT16;
//Materialize buffers
uint64_t* d_MAT_IDX17;
hipMalloc(&d_MAT_IDX17, sizeof(uint64_t));
hipMemset(d_MAT_IDX17, 0, sizeof(uint64_t));
auto MAT17aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT17);
DBDecimalType* d_MAT17aggr0__tmp_attr0;
hipMalloc(&d_MAT17aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT17);
main_18<<<std::ceil((float)COUNT16/(float)TILE_SIZE), TB>>>(COUNT16, d_MAT17aggr0__tmp_attr0, d_MAT_IDX17, d_aggr0__tmp_attr0, d_cycles_per_warp_main_18_materialize_17);
uint64_t MATCOUNT_17 = 0;
hipMemcpy(&MATCOUNT_17, d_MAT_IDX17, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT17aggr0__tmp_attr0, d_MAT17aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT17, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_18_materialize_17 = (int64_t*)malloc(sizeof(int64_t) * main_18_materialize_17_cpw_size);
hipMemcpy(cycles_per_warp_main_18_materialize_17, d_cycles_per_warp_main_18_materialize_17, sizeof(int64_t) * main_18_materialize_17_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_18_materialize_17 ";
for (auto i=0ull; i < main_18_materialize_17_cpw_size; i++) std::cout << cycles_per_warp_main_18_materialize_17[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_14);
hipFree(d_BUF_IDX_14);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_MAT17aggr0__tmp_attr0);
hipFree(d_MAT_IDX17);
free(MAT17aggr0__tmp_attr0);
}