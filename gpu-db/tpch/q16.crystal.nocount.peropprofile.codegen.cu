#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_INSERT HT_6, int64_t* cycles_per_warp_main_1_join_build_6, int64_t* cycles_per_warp_main_1_selection_0, int64_t* cycles_per_warp_main_1_selection_2, int64_t* cycles_per_warp_main_1_selection_3, DBStringType* part__p_brand, DBI32Type* part__p_partkey, DBI32Type* part__p_size, DBStringType* part__p_type, size_t part_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBI32Type reg_part__p_size[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_size[ITEM] = part__p_size[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_part__p_size[ITEM], 49, Predicate::eq)) || (evaluatePredicate(reg_part__p_size[ITEM], 14, Predicate::eq)) || (evaluatePredicate(reg_part__p_size[ITEM], 23, Predicate::eq)) || (evaluatePredicate(reg_part__p_size[ITEM], 45, Predicate::eq)) || (evaluatePredicate(reg_part__p_size[ITEM], 19, Predicate::eq)) || (evaluatePredicate(reg_part__p_size[ITEM], 3, Predicate::eq)) || (evaluatePredicate(reg_part__p_size[ITEM], 36, Predicate::eq)) || (evaluatePredicate(reg_part__p_size[ITEM], 9, Predicate::eq));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
DBStringType reg_part__p_brand[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_brand[ITEM] = part__p_brand[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_part__p_brand[ITEM], "Brand#45", Predicate::neq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
DBStringType reg_part__p_type[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_type[ITEM] = part__p_type[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(Like(reg_part__p_type[ITEM], "MEDIUM POLISHED", "", nullptr, nullptr, 0));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_3[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_6.insert(cuco::pair{KEY_6[ITEM], ITEM*TB + tid});
BUF_6[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_build_6[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT_SJ>
__global__ void main_5(HASHTABLE_INSERT_SJ HT_8, int64_t* cycles_per_warp_main_5_anti_semi_join_build_8, int64_t* cycles_per_warp_main_5_selection_4, DBStringType* supplier__s_comment, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_supplier__s_comment[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_comment[ITEM] = supplier__s_comment[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= Like(reg_supplier__s_comment[ITEM], "", "", (const char*[]){ "Customer", "Complaints" }, (const int[]){ 8, 10 }, 2);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_selection_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_8.insert(cuco::pair{KEY_8[ITEM], 1});
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_anti_semi_join_build_8[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_PROBE_SJ, typename HASHTABLE_FIND>
__global__ void main_7(uint64_t* BUF_6, HASHTABLE_PROBE HT_6, HASHTABLE_PROBE_SJ HT_8, HASHTABLE_FIND HT_9, DBI16Type* KEY_9part__p_brand_encoded, DBI32Type* KEY_9part__p_size, DBI16Type* KEY_9part__p_type_encoded, int* SLOT_COUNT_9, DBI64Type* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_7_aggregation_9, int64_t* cycles_per_warp_main_7_anti_semi_join_probe_8, int64_t* cycles_per_warp_main_7_join_probe_6, DBI16Type* part__p_brand_encoded, DBI32Type* part__p_size, DBI16Type* part__p_type_encoded, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_partsupp__ps_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
reg_partsupp__ps_partkey[ITEM] = partsupp__ps_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_partsupp__ps_partkey[ITEM];
}
//Probe Hash table
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_probe_6[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_partsupp__ps_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
reg_partsupp__ps_suppkey[ITEM] = partsupp__ps_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_partsupp__ps_suppkey[ITEM];
}
//Probe Hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_8 = HT_8.find(KEY_8[ITEM]);
if (!(SLOT_8 == HT_8.end())) {selection_flags[ITEM] = 0;}
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_anti_semi_join_probe_8[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_9[ITEMS_PER_THREAD];
DBI16Type reg_part__p_brand_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_brand_encoded[ITEM] = part__p_brand_encoded[BUF_6[slot_second6[ITEM] * 1 + 0]];
}
DBI16Type reg_part__p_type_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_type_encoded[ITEM] = part__p_type_encoded[BUF_6[slot_second6[ITEM] * 1 + 0]];
}
DBI32Type reg_part__p_size[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_size[ITEM] = part__p_size[BUF_6[slot_second6[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_9[ITEM] = 0;
KEY_9[ITEM] |= reg_part__p_brand_encoded[ITEM];
KEY_9[ITEM] <<= 16;
KEY_9[ITEM] |= reg_part__p_type_encoded[ITEM];
KEY_9[ITEM] <<= 32;
KEY_9[ITEM] |= reg_part__p_size[ITEM];
}
//Aggregate in hashtable
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_9 = get_aggregation_slot(KEY_9[ITEM], HT_9, SLOT_COUNT_9);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_9], 1);
KEY_9part__p_brand_encoded[buf_idx_9] = reg_part__p_brand_encoded[ITEM];
KEY_9part__p_type_encoded[buf_idx_9] = reg_part__p_type_encoded[ITEM];
KEY_9part__p_size[buf_idx_9] = reg_part__p_size[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_aggregation_9[blockIdx.x] = cycles_per_warp;}
}
__global__ void main_11(size_t COUNT9, DBI64Type* MAT10aggr0__tmp_attr0, DBI16Type* MAT10part__p_brand_encoded, DBI32Type* MAT10part__p_size, DBI16Type* MAT10part__p_type_encoded, uint64_t* MAT_IDX10, DBI64Type* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_11_materialize_10, DBI16Type* part__p_brand_encoded, DBI32Type* part__p_size, DBI16Type* part__p_type_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBI16Type reg_part__p_brand_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
reg_part__p_brand_encoded[ITEM] = part__p_brand_encoded[ITEM*TB + tid];
}
DBI16Type reg_part__p_type_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
reg_part__p_type_encoded[ITEM] = part__p_type_encoded[ITEM*TB + tid];
}
DBI32Type reg_part__p_size[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
reg_part__p_size[ITEM] = part__p_size[ITEM*TB + tid];
}
DBI64Type reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx10 = atomicAdd((int*)MAT_IDX10, 1);
MAT10part__p_brand_encoded[mat_idx10] = reg_part__p_brand_encoded[ITEM];
MAT10part__p_type_encoded[mat_idx10] = reg_part__p_type_encoded[ITEM];
MAT10part__p_size[mat_idx10] = reg_part__p_size[ITEM];
MAT10aggr0__tmp_attr0[mat_idx10] = reg_aggr0__tmp_attr0[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_materialize_10[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_2;
auto main_1_selection_2_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_2, -1, sizeof(int64_t) * main_1_selection_2_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_3;
auto main_1_selection_3_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_3, -1, sizeof(int64_t) * main_1_selection_3_cpw_size);
int64_t* d_cycles_per_warp_main_1_join_build_6;
auto main_1_join_build_6_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_build_6, sizeof(int64_t) * main_1_join_build_6_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_build_6, -1, sizeof(int64_t) * main_1_join_build_6_cpw_size);
size_t COUNT6 = part_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 1);
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)part_size/(float)TILE_SIZE), TB>>>(d_BUF_6, d_BUF_IDX_6, d_HT_6.ref(cuco::insert), d_cycles_per_warp_main_1_join_build_6, d_cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_3, d_part__p_brand, d_part__p_partkey, d_part__p_size, d_part__p_type, part_size);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_2 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_2 ";
for (auto i=0ull; i < main_1_selection_2_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_3 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_3, d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_3 ";
for (auto i=0ull; i < main_1_selection_3_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_3[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_build_6 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_build_6_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_build_6, d_cycles_per_warp_main_1_join_build_6, sizeof(int64_t) * main_1_join_build_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_build_6 ";
for (auto i=0ull; i < main_1_join_build_6_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_build_6[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_5_selection_4;
auto main_5_selection_4_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_selection_4, sizeof(int64_t) * main_5_selection_4_cpw_size);
hipMemset(d_cycles_per_warp_main_5_selection_4, -1, sizeof(int64_t) * main_5_selection_4_cpw_size);
int64_t* d_cycles_per_warp_main_5_anti_semi_join_build_8;
auto main_5_anti_semi_join_build_8_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_anti_semi_join_build_8, sizeof(int64_t) * main_5_anti_semi_join_build_8_cpw_size);
hipMemset(d_cycles_per_warp_main_5_anti_semi_join_build_8, -1, sizeof(int64_t) * main_5_anti_semi_join_build_8_cpw_size);
size_t COUNT8 = supplier_size;
// Insert hash table control;
auto d_HT_8 = cuco::static_map{ (int)COUNT8*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TB>>>(d_HT_8.ref(cuco::insert), d_cycles_per_warp_main_5_anti_semi_join_build_8, d_cycles_per_warp_main_5_selection_4, d_supplier__s_comment, d_supplier__s_suppkey, supplier_size);
int64_t* d_cycles_per_warp_main_7_join_probe_6;
auto main_7_join_probe_6_cpw_size = std::ceil((float)partsupp_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_probe_6, sizeof(int64_t) * main_7_join_probe_6_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_probe_6, -1, sizeof(int64_t) * main_7_join_probe_6_cpw_size);
int64_t* d_cycles_per_warp_main_7_anti_semi_join_probe_8;
auto main_7_anti_semi_join_probe_8_cpw_size = std::ceil((float)partsupp_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_anti_semi_join_probe_8, sizeof(int64_t) * main_7_anti_semi_join_probe_8_cpw_size);
hipMemset(d_cycles_per_warp_main_7_anti_semi_join_probe_8, -1, sizeof(int64_t) * main_7_anti_semi_join_probe_8_cpw_size);
int64_t* d_cycles_per_warp_main_7_aggregation_9;
auto main_7_aggregation_9_cpw_size = std::ceil((float)partsupp_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_aggregation_9, sizeof(int64_t) * main_7_aggregation_9_cpw_size);
hipMemset(d_cycles_per_warp_main_7_aggregation_9, -1, sizeof(int64_t) * main_7_aggregation_9_cpw_size);
size_t COUNT9 = 120976;
auto d_HT_9 = cuco::static_map{ (int)120976*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_9;
hipMalloc(&d_SLOT_COUNT_9, sizeof(int));
hipMemset(d_SLOT_COUNT_9, 0, sizeof(int));
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT9);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT9);
DBI16Type* d_KEY_9part__p_brand_encoded;
hipMalloc(&d_KEY_9part__p_brand_encoded, sizeof(DBI16Type) * COUNT9);
hipMemset(d_KEY_9part__p_brand_encoded, 0, sizeof(DBI16Type) * COUNT9);
DBI16Type* d_KEY_9part__p_type_encoded;
hipMalloc(&d_KEY_9part__p_type_encoded, sizeof(DBI16Type) * COUNT9);
hipMemset(d_KEY_9part__p_type_encoded, 0, sizeof(DBI16Type) * COUNT9);
DBI32Type* d_KEY_9part__p_size;
hipMalloc(&d_KEY_9part__p_size, sizeof(DBI32Type) * COUNT9);
hipMemset(d_KEY_9part__p_size, 0, sizeof(DBI32Type) * COUNT9);
main_7<<<std::ceil((float)partsupp_size/(float)TILE_SIZE), TB>>>(d_BUF_6, d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::find), d_HT_9.ref(cuco::insert_and_find), d_KEY_9part__p_brand_encoded, d_KEY_9part__p_size, d_KEY_9part__p_type_encoded, d_SLOT_COUNT_9, d_aggr0__tmp_attr0, d_cycles_per_warp_main_7_aggregation_9, d_cycles_per_warp_main_7_anti_semi_join_probe_8, d_cycles_per_warp_main_7_join_probe_6, d_part__p_brand_encoded, d_part__p_size, d_part__p_type_encoded, d_partsupp__ps_partkey, d_partsupp__ps_suppkey, partsupp_size);
int64_t* cycles_per_warp_main_7_join_probe_6 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_probe_6_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_probe_6, d_cycles_per_warp_main_7_join_probe_6, sizeof(int64_t) * main_7_join_probe_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_probe_6 ";
for (auto i=0ull; i < main_7_join_probe_6_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_probe_6[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_anti_semi_join_probe_8 = (int64_t*)malloc(sizeof(int64_t) * main_7_anti_semi_join_probe_8_cpw_size);
hipMemcpy(cycles_per_warp_main_7_anti_semi_join_probe_8, d_cycles_per_warp_main_7_anti_semi_join_probe_8, sizeof(int64_t) * main_7_anti_semi_join_probe_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_anti_semi_join_probe_8 ";
for (auto i=0ull; i < main_7_anti_semi_join_probe_8_cpw_size; i++) std::cout << cycles_per_warp_main_7_anti_semi_join_probe_8[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_aggregation_9 = (int64_t*)malloc(sizeof(int64_t) * main_7_aggregation_9_cpw_size);
hipMemcpy(cycles_per_warp_main_7_aggregation_9, d_cycles_per_warp_main_7_aggregation_9, sizeof(int64_t) * main_7_aggregation_9_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_aggregation_9 ";
for (auto i=0ull; i < main_7_aggregation_9_cpw_size; i++) std::cout << cycles_per_warp_main_7_aggregation_9[i] << " ";
std::cout << std::endl;
COUNT9 = d_HT_9.size();
int64_t* d_cycles_per_warp_main_11_materialize_10;
auto main_11_materialize_10_cpw_size = std::ceil((float)COUNT9/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_materialize_10, sizeof(int64_t) * main_11_materialize_10_cpw_size);
hipMemset(d_cycles_per_warp_main_11_materialize_10, -1, sizeof(int64_t) * main_11_materialize_10_cpw_size);
size_t COUNT10 = COUNT9;
//Materialize buffers
uint64_t* d_MAT_IDX10;
hipMalloc(&d_MAT_IDX10, sizeof(uint64_t));
hipMemset(d_MAT_IDX10, 0, sizeof(uint64_t));
auto MAT10part__p_brand_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT10);
DBI16Type* d_MAT10part__p_brand_encoded;
hipMalloc(&d_MAT10part__p_brand_encoded, sizeof(DBI16Type) * COUNT10);
auto MAT10part__p_type_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT10);
DBI16Type* d_MAT10part__p_type_encoded;
hipMalloc(&d_MAT10part__p_type_encoded, sizeof(DBI16Type) * COUNT10);
auto MAT10part__p_size = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT10);
DBI32Type* d_MAT10part__p_size;
hipMalloc(&d_MAT10part__p_size, sizeof(DBI32Type) * COUNT10);
auto MAT10aggr0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT10);
DBI64Type* d_MAT10aggr0__tmp_attr0;
hipMalloc(&d_MAT10aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT10);
main_11<<<std::ceil((float)COUNT9/(float)TILE_SIZE), TB>>>(COUNT9, d_MAT10aggr0__tmp_attr0, d_MAT10part__p_brand_encoded, d_MAT10part__p_size, d_MAT10part__p_type_encoded, d_MAT_IDX10, d_aggr0__tmp_attr0, d_cycles_per_warp_main_11_materialize_10, d_KEY_9part__p_brand_encoded, d_KEY_9part__p_size, d_KEY_9part__p_type_encoded);
uint64_t MATCOUNT_10 = 0;
hipMemcpy(&MATCOUNT_10, d_MAT_IDX10, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT10part__p_brand_encoded, d_MAT10part__p_brand_encoded, sizeof(DBI16Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10part__p_type_encoded, d_MAT10part__p_type_encoded, sizeof(DBI16Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10part__p_size, d_MAT10part__p_size, sizeof(DBI32Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10aggr0__tmp_attr0, d_MAT10aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT10, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_11_materialize_10 = (int64_t*)malloc(sizeof(int64_t) * main_11_materialize_10_cpw_size);
hipMemcpy(cycles_per_warp_main_11_materialize_10, d_cycles_per_warp_main_11_materialize_10, sizeof(int64_t) * main_11_materialize_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_materialize_10 ";
for (auto i=0ull; i < main_11_materialize_10_cpw_size; i++) std::cout << cycles_per_warp_main_11_materialize_10[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_KEY_9part__p_brand_encoded);
hipFree(d_KEY_9part__p_size);
hipFree(d_KEY_9part__p_type_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_MAT10aggr0__tmp_attr0);
hipFree(d_MAT10part__p_brand_encoded);
hipFree(d_MAT10part__p_size);
hipFree(d_MAT10part__p_type_encoded);
hipFree(d_MAT_IDX10);
free(MAT10aggr0__tmp_attr0);
free(MAT10part__p_brand_encoded);
free(MAT10part__p_size);
free(MAT10part__p_type_encoded);
}