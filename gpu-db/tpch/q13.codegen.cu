#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_1(uint64_t* COUNT0, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
//Materialize count
atomicAdd((int*)COUNT0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0, buf_idx_0});
BUF_0[buf_idx_0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_3(uint64_t* BUF_0, HASHTABLE_PROBE HT_0, HASHTABLE_INSERT HT_2, DBI32Type* customer__c_custkey, DBI32Type* orders__o_custkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_0 |= reg_orders__o_custkey;
//Probe Hash table
HT_0.for_each(KEY_0, [&] __device__ (auto const SLOT_0) {

auto const [slot_first0, slot_second0] = SLOT_0;
if (!(true)) return;
uint64_t KEY_2 = 0;
auto reg_customer__c_custkey = customer__c_custkey[BUF_0[slot_second0 * 1 + 0]];

KEY_2 |= reg_customer__c_custkey;
//Create aggregation hash table
HT_2.insert(cuco::pair{KEY_2, 1});
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_3(uint64_t* BUF_0, HASHTABLE_PROBE HT_0, HASHTABLE_FIND HT_2, DBI32Type* KEY_2customer__c_custkey, DBI64Type* aggr0__tmp_attr0, DBI32Type* customer__c_custkey, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_0 |= reg_orders__o_custkey;
//Probe Hash table
HT_0.for_each(KEY_0, [&] __device__ (auto const SLOT_0) {
auto const [slot_first0, slot_second0] = SLOT_0;
if (!(true)) return;
uint64_t KEY_2 = 0;
auto reg_customer__c_custkey = customer__c_custkey[BUF_0[slot_second0 * 1 + 0]];

KEY_2 |= reg_customer__c_custkey;
//Aggregate in hashtable
auto buf_idx_2 = HT_2.find(KEY_2)->second;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_2], 1);
KEY_2customer__c_custkey[buf_idx_2] = reg_customer__c_custkey;
});
}
template<typename HASHTABLE_INSERT>
__global__ void count_5(size_t COUNT2, HASHTABLE_INSERT HT_4, DBI64Type* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT2) return;
uint64_t KEY_4 = 0;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];

KEY_4 |= (DBI32Type)reg_aggr0__tmp_attr0;
//Create aggregation hash table
HT_4.insert(cuco::pair{KEY_4, 1});
}
template<typename HASHTABLE_FIND>
__global__ void main_5(size_t COUNT2, HASHTABLE_FIND HT_4, DBI64Type* KEY_4aggr0__tmp_attr0, DBI64Type* aggr0__tmp_attr0, DBI64Type* aggr1__tmp_attr1) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT2) return;
uint64_t KEY_4 = 0;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];

KEY_4 |= (DBI32Type)reg_aggr0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_4 = HT_4.find(KEY_4)->second;
aggregate_sum(&aggr1__tmp_attr1[buf_idx_4], 1);
KEY_4aggr0__tmp_attr0[buf_idx_4] = reg_aggr0__tmp_attr0;
}
__global__ void count_7(size_t COUNT4, uint64_t* COUNT6) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT4) return;
//Materialize count
atomicAdd((int*)COUNT6, 1);
}
__global__ void main_7(size_t COUNT4, DBI64Type* MAT6aggr0__tmp_attr0, DBI64Type* MAT6aggr1__tmp_attr1, uint64_t* MAT_IDX6, DBI64Type* aggr0__tmp_attr0, DBI64Type* aggr1__tmp_attr1) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT4) return;
//Materialize buffers
auto mat_idx6 = atomicAdd((int*)MAT_IDX6, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT6aggr0__tmp_attr0[mat_idx6] = reg_aggr0__tmp_attr0;
auto reg_aggr1__tmp_attr1 = aggr1__tmp_attr1[tid];
MAT6aggr1__tmp_attr1[mat_idx6] = reg_aggr1__tmp_attr1;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)customer_size/128.), 128>>>(d_COUNT0, customer_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::experimental::static_multimap{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_customer__c_custkey, customer_size);
//Create aggregation hash table
auto d_HT_2 = cuco::static_map{ (int)1500000*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_3<<<std::ceil((float)orders_size/128.), 128>>>(d_BUF_0, d_HT_0.ref(cuco::for_each), d_HT_2.ref(cuco::insert), d_customer__c_custkey, d_orders__o_custkey, orders_size);
size_t COUNT2 = d_HT_2.size();
thrust::device_vector<int64_t> keys_2(COUNT2), vals_2(COUNT2);
d_HT_2.retrieve_all(keys_2.begin(), vals_2.begin());
d_HT_2.clear();
int64_t* raw_keys2 = thrust::raw_pointer_cast(keys_2.data());
insertKeys<<<std::ceil((float)COUNT2/128.), 128>>>(raw_keys2, d_HT_2.ref(cuco::insert), COUNT2);
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT2);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT2);
DBI32Type* d_KEY_2customer__c_custkey;
hipMalloc(&d_KEY_2customer__c_custkey, sizeof(DBI32Type) * COUNT2);
hipMemset(d_KEY_2customer__c_custkey, 0, sizeof(DBI32Type) * COUNT2);
main_3<<<std::ceil((float)orders_size/128.), 128>>>(d_BUF_0, d_HT_0.ref(cuco::for_each), d_HT_2.ref(cuco::find), d_KEY_2customer__c_custkey, d_aggr0__tmp_attr0, d_customer__c_custkey, d_orders__o_custkey, d_orders__o_orderkey, orders_size);
//Create aggregation hash table
auto d_HT_4 = cuco::static_map{ (int)1500000*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5<<<std::ceil((float)COUNT2/128.), 128>>>(COUNT2, d_HT_4.ref(cuco::insert), d_aggr0__tmp_attr0);
size_t COUNT4 = d_HT_4.size();
thrust::device_vector<int64_t> keys_4(COUNT4), vals_4(COUNT4);
d_HT_4.retrieve_all(keys_4.begin(), vals_4.begin());
d_HT_4.clear();
int64_t* raw_keys4 = thrust::raw_pointer_cast(keys_4.data());
insertKeys<<<std::ceil((float)COUNT4/128.), 128>>>(raw_keys4, d_HT_4.ref(cuco::insert), COUNT4);
//Aggregate in hashtable
DBI64Type* d_aggr1__tmp_attr1;
hipMalloc(&d_aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT4);
hipMemset(d_aggr1__tmp_attr1, 0, sizeof(DBI64Type) * COUNT4);
DBI64Type* d_KEY_4aggr0__tmp_attr0;
hipMalloc(&d_KEY_4aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT4);
hipMemset(d_KEY_4aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT4);
main_5<<<std::ceil((float)COUNT2/128.), 128>>>(COUNT2, d_HT_4.ref(cuco::find), d_KEY_4aggr0__tmp_attr0, d_aggr0__tmp_attr0, d_aggr1__tmp_attr1);
//Materialize count
uint64_t* d_COUNT6;
hipMalloc(&d_COUNT6, sizeof(uint64_t));
hipMemset(d_COUNT6, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)COUNT4/128.), 128>>>(COUNT4, d_COUNT6);
uint64_t COUNT6;
hipMemcpy(&COUNT6, d_COUNT6, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX6;
hipMalloc(&d_MAT_IDX6, sizeof(uint64_t));
hipMemset(d_MAT_IDX6, 0, sizeof(uint64_t));
auto MAT6aggr0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT6);
DBI64Type* d_MAT6aggr0__tmp_attr0;
hipMalloc(&d_MAT6aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT6);
auto MAT6aggr1__tmp_attr1 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT6);
DBI64Type* d_MAT6aggr1__tmp_attr1;
hipMalloc(&d_MAT6aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT6);
main_7<<<std::ceil((float)COUNT4/128.), 128>>>(COUNT4, d_MAT6aggr0__tmp_attr0, d_MAT6aggr1__tmp_attr1, d_MAT_IDX6, d_KEY_4aggr0__tmp_attr0, d_aggr1__tmp_attr1);
hipMemcpy(MAT6aggr0__tmp_attr0, d_MAT6aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT6, hipMemcpyDeviceToHost);
hipMemcpy(MAT6aggr1__tmp_attr1, d_MAT6aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT6, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT6; i++) { std::cout << "" << MAT6aggr0__tmp_attr0[i];
std::cout << "|" << MAT6aggr1__tmp_attr1[i];
std::cout << std::endl; }
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_COUNT0);
hipFree(d_KEY_2customer__c_custkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_KEY_4aggr0__tmp_attr0);
hipFree(d_aggr1__tmp_attr1);
hipFree(d_COUNT6);
hipFree(d_MAT6aggr0__tmp_attr0);
hipFree(d_MAT6aggr1__tmp_attr1);
hipFree(d_MAT_IDX6);
free(MAT6aggr0__tmp_attr0);
free(MAT6aggr1__tmp_attr1);
}