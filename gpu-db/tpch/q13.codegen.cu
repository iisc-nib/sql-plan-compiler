#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_55c94de8ff80(uint64_t* COUNT55c94de8ba50, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
//Materialize count
atomicAdd((int*)COUNT55c94de8ba50, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_55c94de8ff80(uint64_t* BUF_55c94de8ba50, uint64_t* BUF_IDX_55c94de8ba50, HASHTABLE_INSERT HT_55c94de8ba50, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_55c94de8ba50 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_55c94de8ba50 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_55c94de8ba50 = atomicAdd((int*)BUF_IDX_55c94de8ba50, 1);
HT_55c94de8ba50.insert(cuco::pair{KEY_55c94de8ba50, buf_idx_55c94de8ba50});
BUF_55c94de8ba50[buf_idx_55c94de8ba50 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_55c94de91740(uint64_t* BUF_55c94de8ba50, HASHTABLE_INSERT HT_55c94de49740, HASHTABLE_PROBE HT_55c94de8ba50, DBI32Type* customer__c_custkey, DBI32Type* orders__o_custkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_55c94de8ba50 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_55c94de8ba50 |= reg_orders__o_custkey;
//Probe Hash table
HT_55c94de8ba50.for_each(KEY_55c94de8ba50, [&] __device__ (auto const SLOT_55c94de8ba50) {

auto const [slot_first55c94de8ba50, slot_second55c94de8ba50] = SLOT_55c94de8ba50;
if (!(true)) return;
uint64_t KEY_55c94de49740 = 0;
auto reg_customer__c_custkey = customer__c_custkey[BUF_55c94de8ba50[slot_second55c94de8ba50 * 1 + 0]];

KEY_55c94de49740 |= reg_customer__c_custkey;
//Create aggregation hash table
HT_55c94de49740.insert(cuco::pair{KEY_55c94de49740, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_55c94de91740(uint64_t* BUF_55c94de8ba50, HASHTABLE_FIND HT_55c94de49740, HASHTABLE_PROBE HT_55c94de8ba50, DBI32Type* KEY_55c94de49740customer__c_custkey, DBI64Type* aggr0__tmp_attr0, DBI32Type* customer__c_custkey, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_55c94de8ba50 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_55c94de8ba50 |= reg_orders__o_custkey;
//Probe Hash table
HT_55c94de8ba50.for_each(KEY_55c94de8ba50, [&] __device__ (auto const SLOT_55c94de8ba50) {
auto const [slot_first55c94de8ba50, slot_second55c94de8ba50] = SLOT_55c94de8ba50;
if (!(true)) return;
uint64_t KEY_55c94de49740 = 0;
auto reg_customer__c_custkey = customer__c_custkey[BUF_55c94de8ba50[slot_second55c94de8ba50 * 1 + 0]];

KEY_55c94de49740 |= reg_customer__c_custkey;
//Aggregate in hashtable
auto buf_idx_55c94de49740 = HT_55c94de49740.find(KEY_55c94de49740)->second;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_55c94de49740], 1);
KEY_55c94de49740customer__c_custkey[buf_idx_55c94de49740] = reg_customer__c_custkey;
});
}
template<typename HASHTABLE_INSERT>
__global__ void count_55c94de98040(size_t COUNT55c94de49740, HASHTABLE_INSERT HT_55c94de4a1d0, DBI64Type* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT55c94de49740) return;
uint64_t KEY_55c94de4a1d0 = 0;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];

KEY_55c94de4a1d0 |= (DBI32Type)reg_aggr0__tmp_attr0;
//Create aggregation hash table
HT_55c94de4a1d0.insert(cuco::pair{KEY_55c94de4a1d0, 1});
}
template<typename HASHTABLE_FIND>
__global__ void main_55c94de98040(size_t COUNT55c94de49740, HASHTABLE_FIND HT_55c94de4a1d0, DBI64Type* KEY_55c94de4a1d0aggr0__tmp_attr0, DBI64Type* aggr0__tmp_attr0, DBI64Type* aggr1__tmp_attr1) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT55c94de49740) return;
uint64_t KEY_55c94de4a1d0 = 0;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];

KEY_55c94de4a1d0 |= (DBI32Type)reg_aggr0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_55c94de4a1d0 = HT_55c94de4a1d0.find(KEY_55c94de4a1d0)->second;
aggregate_sum(&aggr1__tmp_attr1[buf_idx_55c94de4a1d0], 1);
KEY_55c94de4a1d0aggr0__tmp_attr0[buf_idx_55c94de4a1d0] = reg_aggr0__tmp_attr0;
}
__global__ void count_55c94de99a50(size_t COUNT55c94de4a1d0, uint64_t* COUNT55c94de5db70) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT55c94de4a1d0) return;
//Materialize count
atomicAdd((int*)COUNT55c94de5db70, 1);
}
__global__ void main_55c94de99a50(size_t COUNT55c94de4a1d0, DBI64Type* MAT55c94de5db70aggr0__tmp_attr0, DBI64Type* MAT55c94de5db70aggr1__tmp_attr1, uint64_t* MAT_IDX55c94de5db70, DBI64Type* aggr0__tmp_attr0, DBI64Type* aggr1__tmp_attr1) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT55c94de4a1d0) return;
//Materialize buffers
auto mat_idx55c94de5db70 = atomicAdd((int*)MAT_IDX55c94de5db70, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT55c94de5db70aggr0__tmp_attr0[mat_idx55c94de5db70] = reg_aggr0__tmp_attr0;
auto reg_aggr1__tmp_attr1 = aggr1__tmp_attr1[tid];
MAT55c94de5db70aggr1__tmp_attr1[mat_idx55c94de5db70] = reg_aggr1__tmp_attr1;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map) {
//Materialize count
uint64_t* d_COUNT55c94de8ba50;
hipMalloc(&d_COUNT55c94de8ba50, sizeof(uint64_t));
hipMemset(d_COUNT55c94de8ba50, 0, sizeof(uint64_t));
count_55c94de8ff80<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT55c94de8ba50, customer_size);
uint64_t COUNT55c94de8ba50;
hipMemcpy(&COUNT55c94de8ba50, d_COUNT55c94de8ba50, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_55c94de8ba50;
hipMalloc(&d_BUF_IDX_55c94de8ba50, sizeof(uint64_t));
hipMemset(d_BUF_IDX_55c94de8ba50, 0, sizeof(uint64_t));
uint64_t* d_BUF_55c94de8ba50;
hipMalloc(&d_BUF_55c94de8ba50, sizeof(uint64_t) * COUNT55c94de8ba50 * 1);
auto d_HT_55c94de8ba50 = cuco::experimental::static_multimap{ (int)COUNT55c94de8ba50*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_55c94de8ff80<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_55c94de8ba50, d_BUF_IDX_55c94de8ba50, d_HT_55c94de8ba50.ref(cuco::insert), d_customer__c_custkey, customer_size);
//Create aggregation hash table
auto d_HT_55c94de49740 = cuco::static_map{ (int)1500000*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_55c94de91740<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_55c94de8ba50, d_HT_55c94de49740.ref(cuco::insert), d_HT_55c94de8ba50.ref(cuco::for_each), d_customer__c_custkey, d_orders__o_custkey, orders_size);
size_t COUNT55c94de49740 = d_HT_55c94de49740.size();
thrust::device_vector<int64_t> keys_55c94de49740(COUNT55c94de49740), vals_55c94de49740(COUNT55c94de49740);
d_HT_55c94de49740.retrieve_all(keys_55c94de49740.begin(), vals_55c94de49740.begin());
d_HT_55c94de49740.clear();
int64_t* raw_keys55c94de49740 = thrust::raw_pointer_cast(keys_55c94de49740.data());
insertKeys<<<std::ceil((float)COUNT55c94de49740/32.), 32>>>(raw_keys55c94de49740, d_HT_55c94de49740.ref(cuco::insert), COUNT55c94de49740);
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT55c94de49740);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT55c94de49740);
DBI32Type* d_KEY_55c94de49740customer__c_custkey;
hipMalloc(&d_KEY_55c94de49740customer__c_custkey, sizeof(DBI32Type) * COUNT55c94de49740);
hipMemset(d_KEY_55c94de49740customer__c_custkey, 0, sizeof(DBI32Type) * COUNT55c94de49740);
main_55c94de91740<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_55c94de8ba50, d_HT_55c94de49740.ref(cuco::find), d_HT_55c94de8ba50.ref(cuco::for_each), d_KEY_55c94de49740customer__c_custkey, d_aggr0__tmp_attr0, d_customer__c_custkey, d_orders__o_custkey, d_orders__o_orderkey, orders_size);
//Create aggregation hash table
auto d_HT_55c94de4a1d0 = cuco::static_map{ (int)1500000*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_55c94de98040<<<std::ceil((float)COUNT55c94de49740/32.), 32>>>(COUNT55c94de49740, d_HT_55c94de4a1d0.ref(cuco::insert), d_aggr0__tmp_attr0);
size_t COUNT55c94de4a1d0 = d_HT_55c94de4a1d0.size();
thrust::device_vector<int64_t> keys_55c94de4a1d0(COUNT55c94de4a1d0), vals_55c94de4a1d0(COUNT55c94de4a1d0);
d_HT_55c94de4a1d0.retrieve_all(keys_55c94de4a1d0.begin(), vals_55c94de4a1d0.begin());
d_HT_55c94de4a1d0.clear();
int64_t* raw_keys55c94de4a1d0 = thrust::raw_pointer_cast(keys_55c94de4a1d0.data());
insertKeys<<<std::ceil((float)COUNT55c94de4a1d0/32.), 32>>>(raw_keys55c94de4a1d0, d_HT_55c94de4a1d0.ref(cuco::insert), COUNT55c94de4a1d0);
//Aggregate in hashtable
DBI64Type* d_aggr1__tmp_attr1;
hipMalloc(&d_aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT55c94de4a1d0);
hipMemset(d_aggr1__tmp_attr1, 0, sizeof(DBI64Type) * COUNT55c94de4a1d0);
DBI64Type* d_KEY_55c94de4a1d0aggr0__tmp_attr0;
hipMalloc(&d_KEY_55c94de4a1d0aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT55c94de4a1d0);
hipMemset(d_KEY_55c94de4a1d0aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT55c94de4a1d0);
main_55c94de98040<<<std::ceil((float)COUNT55c94de49740/32.), 32>>>(COUNT55c94de49740, d_HT_55c94de4a1d0.ref(cuco::find), d_KEY_55c94de4a1d0aggr0__tmp_attr0, d_aggr0__tmp_attr0, d_aggr1__tmp_attr1);
//Materialize count
uint64_t* d_COUNT55c94de5db70;
hipMalloc(&d_COUNT55c94de5db70, sizeof(uint64_t));
hipMemset(d_COUNT55c94de5db70, 0, sizeof(uint64_t));
count_55c94de99a50<<<std::ceil((float)COUNT55c94de4a1d0/32.), 32>>>(COUNT55c94de4a1d0, d_COUNT55c94de5db70);
uint64_t COUNT55c94de5db70;
hipMemcpy(&COUNT55c94de5db70, d_COUNT55c94de5db70, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX55c94de5db70;
hipMalloc(&d_MAT_IDX55c94de5db70, sizeof(uint64_t));
hipMemset(d_MAT_IDX55c94de5db70, 0, sizeof(uint64_t));
auto MAT55c94de5db70aggr0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT55c94de5db70);
DBI64Type* d_MAT55c94de5db70aggr0__tmp_attr0;
hipMalloc(&d_MAT55c94de5db70aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT55c94de5db70);
auto MAT55c94de5db70aggr1__tmp_attr1 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT55c94de5db70);
DBI64Type* d_MAT55c94de5db70aggr1__tmp_attr1;
hipMalloc(&d_MAT55c94de5db70aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT55c94de5db70);
main_55c94de99a50<<<std::ceil((float)COUNT55c94de4a1d0/32.), 32>>>(COUNT55c94de4a1d0, d_MAT55c94de5db70aggr0__tmp_attr0, d_MAT55c94de5db70aggr1__tmp_attr1, d_MAT_IDX55c94de5db70, d_KEY_55c94de4a1d0aggr0__tmp_attr0, d_aggr1__tmp_attr1);
hipMemcpy(MAT55c94de5db70aggr0__tmp_attr0, d_MAT55c94de5db70aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT55c94de5db70, hipMemcpyDeviceToHost);
hipMemcpy(MAT55c94de5db70aggr1__tmp_attr1, d_MAT55c94de5db70aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT55c94de5db70, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT55c94de5db70; i++) { std::cout << MAT55c94de5db70aggr0__tmp_attr0[i] << "\t";
std::cout << MAT55c94de5db70aggr1__tmp_attr1[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_55c94de8ba50);
hipFree(d_BUF_IDX_55c94de8ba50);
hipFree(d_COUNT55c94de8ba50);
hipFree(d_KEY_55c94de49740customer__c_custkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_KEY_55c94de4a1d0aggr0__tmp_attr0);
hipFree(d_aggr1__tmp_attr1);
hipFree(d_COUNT55c94de5db70);
hipFree(d_MAT55c94de5db70aggr0__tmp_attr0);
hipFree(d_MAT55c94de5db70aggr1__tmp_attr1);
hipFree(d_MAT_IDX55c94de5db70);
free(MAT55c94de5db70aggr0__tmp_attr0);
free(MAT55c94de5db70aggr1__tmp_attr1);
}