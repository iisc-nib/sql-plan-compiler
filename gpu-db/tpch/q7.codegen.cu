#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_55d9d61ff050(uint64_t* COUNT55d9d61dc060, DBStringType* n1___n_name, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n1___n_name = n1___n_name[tid];
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
//Materialize count
atomicAdd((int*)COUNT55d9d61dc060, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_55d9d61ff050(uint64_t* BUF_55d9d61dc060, uint64_t* BUF_IDX_55d9d61dc060, HASHTABLE_INSERT HT_55d9d61dc060, DBStringType* n1___n_name, DBI32Type* n1___n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n1___n_name = n1___n_name[tid];
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
uint64_t KEY_55d9d61dc060 = 0;
auto reg_n1___n_nationkey = n1___n_nationkey[tid];

KEY_55d9d61dc060 |= reg_n1___n_nationkey;
// Insert hash table kernel;
auto buf_idx_55d9d61dc060 = atomicAdd((int*)BUF_IDX_55d9d61dc060, 1);
HT_55d9d61dc060.insert(cuco::pair{KEY_55d9d61dc060, buf_idx_55d9d61dc060});
BUF_55d9d61dc060[buf_idx_55d9d61dc060 * 1 + 0] = tid;
}
__global__ void count_55d9d6202ac0(uint64_t* COUNT55d9d61fb1f0, DBStringType* n2___n_name, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n2___n_name = n2___n_name[tid];
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
//Materialize count
atomicAdd((int*)COUNT55d9d61fb1f0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_55d9d6202ac0(uint64_t* BUF_55d9d61fb1f0, uint64_t* BUF_IDX_55d9d61fb1f0, HASHTABLE_INSERT HT_55d9d61fb1f0, DBStringType* n2___n_name, DBI32Type* n2___n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n2___n_name = n2___n_name[tid];
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
uint64_t KEY_55d9d61fb1f0 = 0;
auto reg_n2___n_nationkey = n2___n_nationkey[tid];

KEY_55d9d61fb1f0 |= reg_n2___n_nationkey;
// Insert hash table kernel;
auto buf_idx_55d9d61fb1f0 = atomicAdd((int*)BUF_IDX_55d9d61fb1f0, 1);
HT_55d9d61fb1f0.insert(cuco::pair{KEY_55d9d61fb1f0, buf_idx_55d9d61fb1f0});
BUF_55d9d61fb1f0[buf_idx_55d9d61fb1f0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_55d9d6203cb0(uint64_t* BUF_55d9d61fb1f0, uint64_t* COUNT55d9d61f0350, HASHTABLE_PROBE HT_55d9d61fb1f0, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_55d9d61fb1f0 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_55d9d61fb1f0 |= reg_customer__c_nationkey;
//Probe Hash table
HT_55d9d61fb1f0.for_each(KEY_55d9d61fb1f0, [&] __device__ (auto const SLOT_55d9d61fb1f0) {

auto const [slot_first55d9d61fb1f0, slot_second55d9d61fb1f0] = SLOT_55d9d61fb1f0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT55d9d61f0350, 1);
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_55d9d6203cb0(uint64_t* BUF_55d9d61f0350, uint64_t* BUF_55d9d61fb1f0, uint64_t* BUF_IDX_55d9d61f0350, HASHTABLE_INSERT HT_55d9d61f0350, HASHTABLE_PROBE HT_55d9d61fb1f0, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_55d9d61fb1f0 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_55d9d61fb1f0 |= reg_customer__c_nationkey;
//Probe Hash table
HT_55d9d61fb1f0.for_each(KEY_55d9d61fb1f0, [&] __device__ (auto const SLOT_55d9d61fb1f0) {
auto const [slot_first55d9d61fb1f0, slot_second55d9d61fb1f0] = SLOT_55d9d61fb1f0;
if (!(true)) return;
uint64_t KEY_55d9d61f0350 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_55d9d61f0350 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_55d9d61f0350 = atomicAdd((int*)BUF_IDX_55d9d61f0350, 1);
HT_55d9d61f0350.insert(cuco::pair{KEY_55d9d61f0350, buf_idx_55d9d61f0350});
BUF_55d9d61f0350[buf_idx_55d9d61f0350 * 2 + 0] = BUF_55d9d61fb1f0[slot_second55d9d61fb1f0 * 1 + 0];
BUF_55d9d61f0350[buf_idx_55d9d61f0350 * 2 + 1] = tid;
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_55d9d61fd630(uint64_t* BUF_55d9d61f0350, uint64_t* COUNT55d9d61f2090, HASHTABLE_PROBE HT_55d9d61f0350, DBI32Type* orders__o_custkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_55d9d61f0350 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_55d9d61f0350 |= reg_orders__o_custkey;
//Probe Hash table
HT_55d9d61f0350.for_each(KEY_55d9d61f0350, [&] __device__ (auto const SLOT_55d9d61f0350) {

auto const [slot_first55d9d61f0350, slot_second55d9d61f0350] = SLOT_55d9d61f0350;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT55d9d61f2090, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_55d9d61fd630(uint64_t* BUF_55d9d61f0350, uint64_t* BUF_55d9d61f2090, uint64_t* BUF_IDX_55d9d61f2090, HASHTABLE_PROBE HT_55d9d61f0350, HASHTABLE_INSERT HT_55d9d61f2090, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_55d9d61f0350 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_55d9d61f0350 |= reg_orders__o_custkey;
//Probe Hash table
HT_55d9d61f0350.for_each(KEY_55d9d61f0350, [&] __device__ (auto const SLOT_55d9d61f0350) {
auto const [slot_first55d9d61f0350, slot_second55d9d61f0350] = SLOT_55d9d61f0350;
if (!(true)) return;
uint64_t KEY_55d9d61f2090 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_55d9d61f2090 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_55d9d61f2090 = atomicAdd((int*)BUF_IDX_55d9d61f2090, 1);
HT_55d9d61f2090.insert(cuco::pair{KEY_55d9d61f2090, buf_idx_55d9d61f2090});
BUF_55d9d61f2090[buf_idx_55d9d61f2090 * 3 + 0] = tid;
BUF_55d9d61f2090[buf_idx_55d9d61f2090 * 3 + 1] = BUF_55d9d61f0350[slot_second55d9d61f0350 * 2 + 0];
BUF_55d9d61f2090[buf_idx_55d9d61f2090 * 3 + 2] = BUF_55d9d61f0350[slot_second55d9d61f0350 * 2 + 1];
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_55d9d61dae10(uint64_t* BUF_55d9d61dc060, uint64_t* COUNT55d9d61f21a0, HASHTABLE_PROBE HT_55d9d61dc060, DBI32Type* supplier__s_nationkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_55d9d61dc060 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_55d9d61dc060 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_55d9d61dc060.for_each(KEY_55d9d61dc060, [&] __device__ (auto const SLOT_55d9d61dc060) {

auto const [slot_first55d9d61dc060, slot_second55d9d61dc060] = SLOT_55d9d61dc060;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT55d9d61f21a0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_55d9d61dae10(uint64_t* BUF_55d9d61dc060, uint64_t* BUF_55d9d61f21a0, uint64_t* BUF_IDX_55d9d61f21a0, HASHTABLE_PROBE HT_55d9d61dc060, HASHTABLE_INSERT HT_55d9d61f21a0, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_55d9d61dc060 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_55d9d61dc060 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_55d9d61dc060.for_each(KEY_55d9d61dc060, [&] __device__ (auto const SLOT_55d9d61dc060) {
auto const [slot_first55d9d61dc060, slot_second55d9d61dc060] = SLOT_55d9d61dc060;
if (!(true)) return;
uint64_t KEY_55d9d61f21a0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_55d9d61f21a0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_55d9d61f21a0 = atomicAdd((int*)BUF_IDX_55d9d61f21a0, 1);
HT_55d9d61f21a0.insert(cuco::pair{KEY_55d9d61f21a0, buf_idx_55d9d61f21a0});
BUF_55d9d61f21a0[buf_idx_55d9d61f21a0 * 2 + 0] = tid;
BUF_55d9d61f21a0[buf_idx_55d9d61f21a0 * 2 + 1] = BUF_55d9d61dc060[slot_second55d9d61dc060 * 1 + 0];
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_55d9d61db3e0(uint64_t* BUF_55d9d61f2090, uint64_t* BUF_55d9d61f21a0, HASHTABLE_INSERT HT_55d9d61a8de0, HASHTABLE_PROBE HT_55d9d61f2090, HASHTABLE_PROBE HT_55d9d61f21a0, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n1___n_name, DBI16Type* n1___n_name_encoded, DBStringType* n2___n_name, DBI16Type* n2___n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9861, Predicate::lte))) return;
uint64_t KEY_55d9d61f2090 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_55d9d61f2090 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_55d9d61f2090.for_each(KEY_55d9d61f2090, [&] __device__ (auto const SLOT_55d9d61f2090) {

auto const [slot_first55d9d61f2090, slot_second55d9d61f2090] = SLOT_55d9d61f2090;
if (!(true)) return;
uint64_t KEY_55d9d61f21a0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_55d9d61f21a0 |= reg_lineitem__l_suppkey;
//Probe Hash table
HT_55d9d61f21a0.for_each(KEY_55d9d61f21a0, [&] __device__ (auto const SLOT_55d9d61f21a0) {

auto const [slot_first55d9d61f21a0, slot_second55d9d61f21a0] = SLOT_55d9d61f21a0;
auto reg_n1___n_name = n1___n_name[BUF_55d9d61f21a0[slot_second55d9d61f21a0 * 2 + 1]];
auto reg_n2___n_name = n2___n_name[BUF_55d9d61f2090[slot_second55d9d61f2090 * 3 + 1]];
if (!((((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq))) || ((evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) && (true))) return;
uint64_t KEY_55d9d61a8de0 = 0;
auto reg_n1___n_name_encoded = n1___n_name_encoded[BUF_55d9d61f21a0[slot_second55d9d61f21a0 * 2 + 1]];

KEY_55d9d61a8de0 |= reg_n1___n_name_encoded;
auto reg_n2___n_name_encoded = n2___n_name_encoded[BUF_55d9d61f2090[slot_second55d9d61f2090 * 3 + 1]];
KEY_55d9d61a8de0 <<= 16;
KEY_55d9d61a8de0 |= reg_n2___n_name_encoded;
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_lineitem__l_shipdate);
KEY_55d9d61a8de0 <<= 32;
KEY_55d9d61a8de0 |= (DBI32Type)reg_map0__tmp_attr0;
//Create aggregation hash table
HT_55d9d61a8de0.insert(cuco::pair{KEY_55d9d61a8de0, 1});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_55d9d61db3e0(uint64_t* BUF_55d9d61f2090, uint64_t* BUF_55d9d61f21a0, HASHTABLE_FIND HT_55d9d61a8de0, HASHTABLE_PROBE HT_55d9d61f2090, HASHTABLE_PROBE HT_55d9d61f21a0, DBI64Type* KEY_55d9d61a8de0map0__tmp_attr0, DBI16Type* KEY_55d9d61a8de0n1___n_name_encoded, DBI16Type* KEY_55d9d61a8de0n2___n_name_encoded, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n1___n_name, DBI16Type* n1___n_name_encoded, DBStringType* n2___n_name, DBI16Type* n2___n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9861, Predicate::lte))) return;
uint64_t KEY_55d9d61f2090 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_55d9d61f2090 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_55d9d61f2090.for_each(KEY_55d9d61f2090, [&] __device__ (auto const SLOT_55d9d61f2090) {
auto const [slot_first55d9d61f2090, slot_second55d9d61f2090] = SLOT_55d9d61f2090;
if (!(true)) return;
uint64_t KEY_55d9d61f21a0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_55d9d61f21a0 |= reg_lineitem__l_suppkey;
//Probe Hash table
HT_55d9d61f21a0.for_each(KEY_55d9d61f21a0, [&] __device__ (auto const SLOT_55d9d61f21a0) {
auto const [slot_first55d9d61f21a0, slot_second55d9d61f21a0] = SLOT_55d9d61f21a0;
auto reg_n1___n_name = n1___n_name[BUF_55d9d61f21a0[slot_second55d9d61f21a0 * 2 + 1]];
auto reg_n2___n_name = n2___n_name[BUF_55d9d61f2090[slot_second55d9d61f2090 * 3 + 1]];
if (!((((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq))) || ((evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) && (true))) return;
uint64_t KEY_55d9d61a8de0 = 0;
auto reg_n1___n_name_encoded = n1___n_name_encoded[BUF_55d9d61f21a0[slot_second55d9d61f21a0 * 2 + 1]];

KEY_55d9d61a8de0 |= reg_n1___n_name_encoded;
auto reg_n2___n_name_encoded = n2___n_name_encoded[BUF_55d9d61f2090[slot_second55d9d61f2090 * 3 + 1]];
KEY_55d9d61a8de0 <<= 16;
KEY_55d9d61a8de0 |= reg_n2___n_name_encoded;
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_lineitem__l_shipdate);
KEY_55d9d61a8de0 <<= 32;
KEY_55d9d61a8de0 |= (DBI32Type)reg_map0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_55d9d61a8de0 = HT_55d9d61a8de0.find(KEY_55d9d61a8de0)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_55d9d61a8de0], reg_map0__tmp_attr1);
KEY_55d9d61a8de0n1___n_name_encoded[buf_idx_55d9d61a8de0] = reg_n1___n_name_encoded;
KEY_55d9d61a8de0n2___n_name_encoded[buf_idx_55d9d61a8de0] = reg_n2___n_name_encoded;
KEY_55d9d61a8de0map0__tmp_attr0[buf_idx_55d9d61a8de0] = reg_map0__tmp_attr0;
});
});
}
__global__ void count_55d9d6217d40(size_t COUNT55d9d61a8de0, uint64_t* COUNT55d9d61bd2d0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT55d9d61a8de0) return;
//Materialize count
atomicAdd((int*)COUNT55d9d61bd2d0, 1);
}
__global__ void main_55d9d6217d40(size_t COUNT55d9d61a8de0, DBDecimalType* MAT55d9d61bd2d0aggr0__tmp_attr2, DBI64Type* MAT55d9d61bd2d0map0__tmp_attr0, DBI16Type* MAT55d9d61bd2d0n1___n_name_encoded, DBI16Type* MAT55d9d61bd2d0n2___n_name_encoded, uint64_t* MAT_IDX55d9d61bd2d0, DBDecimalType* aggr0__tmp_attr2, DBI64Type* map0__tmp_attr0, DBI16Type* n1___n_name_encoded, DBI16Type* n2___n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT55d9d61a8de0) return;
//Materialize buffers
auto mat_idx55d9d61bd2d0 = atomicAdd((int*)MAT_IDX55d9d61bd2d0, 1);
auto reg_n1___n_name_encoded = n1___n_name_encoded[tid];
MAT55d9d61bd2d0n1___n_name_encoded[mat_idx55d9d61bd2d0] = reg_n1___n_name_encoded;
auto reg_n2___n_name_encoded = n2___n_name_encoded[tid];
MAT55d9d61bd2d0n2___n_name_encoded[mat_idx55d9d61bd2d0] = reg_n2___n_name_encoded;
auto reg_map0__tmp_attr0 = map0__tmp_attr0[tid];
MAT55d9d61bd2d0map0__tmp_attr0[mat_idx55d9d61bd2d0] = reg_map0__tmp_attr0;
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT55d9d61bd2d0aggr0__tmp_attr2[mat_idx55d9d61bd2d0] = reg_aggr0__tmp_attr2;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map) {
//Materialize count
uint64_t* d_COUNT55d9d61dc060;
hipMalloc(&d_COUNT55d9d61dc060, sizeof(uint64_t));
hipMemset(d_COUNT55d9d61dc060, 0, sizeof(uint64_t));
count_55d9d61ff050<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT55d9d61dc060, d_nation__n_name, nation_size);
uint64_t COUNT55d9d61dc060;
hipMemcpy(&COUNT55d9d61dc060, d_COUNT55d9d61dc060, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_55d9d61dc060;
hipMalloc(&d_BUF_IDX_55d9d61dc060, sizeof(uint64_t));
hipMemset(d_BUF_IDX_55d9d61dc060, 0, sizeof(uint64_t));
uint64_t* d_BUF_55d9d61dc060;
hipMalloc(&d_BUF_55d9d61dc060, sizeof(uint64_t) * COUNT55d9d61dc060 * 1);
auto d_HT_55d9d61dc060 = cuco::experimental::static_multimap{ (int)COUNT55d9d61dc060*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_55d9d61ff050<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_55d9d61dc060, d_BUF_IDX_55d9d61dc060, d_HT_55d9d61dc060.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
//Materialize count
uint64_t* d_COUNT55d9d61fb1f0;
hipMalloc(&d_COUNT55d9d61fb1f0, sizeof(uint64_t));
hipMemset(d_COUNT55d9d61fb1f0, 0, sizeof(uint64_t));
count_55d9d6202ac0<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT55d9d61fb1f0, d_nation__n_name, nation_size);
uint64_t COUNT55d9d61fb1f0;
hipMemcpy(&COUNT55d9d61fb1f0, d_COUNT55d9d61fb1f0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_55d9d61fb1f0;
hipMalloc(&d_BUF_IDX_55d9d61fb1f0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_55d9d61fb1f0, 0, sizeof(uint64_t));
uint64_t* d_BUF_55d9d61fb1f0;
hipMalloc(&d_BUF_55d9d61fb1f0, sizeof(uint64_t) * COUNT55d9d61fb1f0 * 1);
auto d_HT_55d9d61fb1f0 = cuco::experimental::static_multimap{ (int)COUNT55d9d61fb1f0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_55d9d6202ac0<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_55d9d61fb1f0, d_BUF_IDX_55d9d61fb1f0, d_HT_55d9d61fb1f0.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
//Materialize count
uint64_t* d_COUNT55d9d61f0350;
hipMalloc(&d_COUNT55d9d61f0350, sizeof(uint64_t));
hipMemset(d_COUNT55d9d61f0350, 0, sizeof(uint64_t));
count_55d9d6203cb0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_55d9d61fb1f0, d_COUNT55d9d61f0350, d_HT_55d9d61fb1f0.ref(cuco::for_each), d_customer__c_nationkey, customer_size);
uint64_t COUNT55d9d61f0350;
hipMemcpy(&COUNT55d9d61f0350, d_COUNT55d9d61f0350, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_55d9d61f0350;
hipMalloc(&d_BUF_IDX_55d9d61f0350, sizeof(uint64_t));
hipMemset(d_BUF_IDX_55d9d61f0350, 0, sizeof(uint64_t));
uint64_t* d_BUF_55d9d61f0350;
hipMalloc(&d_BUF_55d9d61f0350, sizeof(uint64_t) * COUNT55d9d61f0350 * 2);
auto d_HT_55d9d61f0350 = cuco::experimental::static_multimap{ (int)COUNT55d9d61f0350*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_55d9d6203cb0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_55d9d61f0350, d_BUF_55d9d61fb1f0, d_BUF_IDX_55d9d61f0350, d_HT_55d9d61f0350.ref(cuco::insert), d_HT_55d9d61fb1f0.ref(cuco::for_each), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
//Materialize count
uint64_t* d_COUNT55d9d61f2090;
hipMalloc(&d_COUNT55d9d61f2090, sizeof(uint64_t));
hipMemset(d_COUNT55d9d61f2090, 0, sizeof(uint64_t));
count_55d9d61fd630<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_55d9d61f0350, d_COUNT55d9d61f2090, d_HT_55d9d61f0350.ref(cuco::for_each), d_orders__o_custkey, orders_size);
uint64_t COUNT55d9d61f2090;
hipMemcpy(&COUNT55d9d61f2090, d_COUNT55d9d61f2090, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_55d9d61f2090;
hipMalloc(&d_BUF_IDX_55d9d61f2090, sizeof(uint64_t));
hipMemset(d_BUF_IDX_55d9d61f2090, 0, sizeof(uint64_t));
uint64_t* d_BUF_55d9d61f2090;
hipMalloc(&d_BUF_55d9d61f2090, sizeof(uint64_t) * COUNT55d9d61f2090 * 3);
auto d_HT_55d9d61f2090 = cuco::experimental::static_multimap{ (int)COUNT55d9d61f2090*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_55d9d61fd630<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_55d9d61f0350, d_BUF_55d9d61f2090, d_BUF_IDX_55d9d61f2090, d_HT_55d9d61f0350.ref(cuco::for_each), d_HT_55d9d61f2090.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderkey, orders_size);
//Materialize count
uint64_t* d_COUNT55d9d61f21a0;
hipMalloc(&d_COUNT55d9d61f21a0, sizeof(uint64_t));
hipMemset(d_COUNT55d9d61f21a0, 0, sizeof(uint64_t));
count_55d9d61dae10<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_55d9d61dc060, d_COUNT55d9d61f21a0, d_HT_55d9d61dc060.ref(cuco::for_each), d_supplier__s_nationkey, supplier_size);
uint64_t COUNT55d9d61f21a0;
hipMemcpy(&COUNT55d9d61f21a0, d_COUNT55d9d61f21a0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_55d9d61f21a0;
hipMalloc(&d_BUF_IDX_55d9d61f21a0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_55d9d61f21a0, 0, sizeof(uint64_t));
uint64_t* d_BUF_55d9d61f21a0;
hipMalloc(&d_BUF_55d9d61f21a0, sizeof(uint64_t) * COUNT55d9d61f21a0 * 2);
auto d_HT_55d9d61f21a0 = cuco::experimental::static_multimap{ (int)COUNT55d9d61f21a0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_55d9d61dae10<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_55d9d61dc060, d_BUF_55d9d61f21a0, d_BUF_IDX_55d9d61f21a0, d_HT_55d9d61dc060.ref(cuco::for_each), d_HT_55d9d61f21a0.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
//Create aggregation hash table
auto d_HT_55d9d61a8de0 = cuco::static_map{ (int)13634*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_55d9d61db3e0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_55d9d61f2090, d_BUF_55d9d61f21a0, d_HT_55d9d61a8de0.ref(cuco::insert), d_HT_55d9d61f2090.ref(cuco::for_each), d_HT_55d9d61f21a0.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_nation__n_name_encoded, d_nation__n_name, d_nation__n_name_encoded);
size_t COUNT55d9d61a8de0 = d_HT_55d9d61a8de0.size();
thrust::device_vector<int64_t> keys_55d9d61a8de0(COUNT55d9d61a8de0), vals_55d9d61a8de0(COUNT55d9d61a8de0);
d_HT_55d9d61a8de0.retrieve_all(keys_55d9d61a8de0.begin(), vals_55d9d61a8de0.begin());
d_HT_55d9d61a8de0.clear();
int64_t* raw_keys55d9d61a8de0 = thrust::raw_pointer_cast(keys_55d9d61a8de0.data());
insertKeys<<<std::ceil((float)COUNT55d9d61a8de0/32.), 32>>>(raw_keys55d9d61a8de0, d_HT_55d9d61a8de0.ref(cuco::insert), COUNT55d9d61a8de0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT55d9d61a8de0);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT55d9d61a8de0);
DBI16Type* d_KEY_55d9d61a8de0n1___n_name_encoded;
hipMalloc(&d_KEY_55d9d61a8de0n1___n_name_encoded, sizeof(DBI16Type) * COUNT55d9d61a8de0);
hipMemset(d_KEY_55d9d61a8de0n1___n_name_encoded, 0, sizeof(DBI16Type) * COUNT55d9d61a8de0);
DBI16Type* d_KEY_55d9d61a8de0n2___n_name_encoded;
hipMalloc(&d_KEY_55d9d61a8de0n2___n_name_encoded, sizeof(DBI16Type) * COUNT55d9d61a8de0);
hipMemset(d_KEY_55d9d61a8de0n2___n_name_encoded, 0, sizeof(DBI16Type) * COUNT55d9d61a8de0);
DBI64Type* d_KEY_55d9d61a8de0map0__tmp_attr0;
hipMalloc(&d_KEY_55d9d61a8de0map0__tmp_attr0, sizeof(DBI64Type) * COUNT55d9d61a8de0);
hipMemset(d_KEY_55d9d61a8de0map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT55d9d61a8de0);
main_55d9d61db3e0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_55d9d61f2090, d_BUF_55d9d61f21a0, d_HT_55d9d61a8de0.ref(cuco::find), d_HT_55d9d61f2090.ref(cuco::for_each), d_HT_55d9d61f21a0.ref(cuco::for_each), d_KEY_55d9d61a8de0map0__tmp_attr0, d_KEY_55d9d61a8de0n1___n_name_encoded, d_KEY_55d9d61a8de0n2___n_name_encoded, d_aggr0__tmp_attr2, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_nation__n_name_encoded, d_nation__n_name, d_nation__n_name_encoded);
//Materialize count
uint64_t* d_COUNT55d9d61bd2d0;
hipMalloc(&d_COUNT55d9d61bd2d0, sizeof(uint64_t));
hipMemset(d_COUNT55d9d61bd2d0, 0, sizeof(uint64_t));
count_55d9d6217d40<<<std::ceil((float)COUNT55d9d61a8de0/32.), 32>>>(COUNT55d9d61a8de0, d_COUNT55d9d61bd2d0);
uint64_t COUNT55d9d61bd2d0;
hipMemcpy(&COUNT55d9d61bd2d0, d_COUNT55d9d61bd2d0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX55d9d61bd2d0;
hipMalloc(&d_MAT_IDX55d9d61bd2d0, sizeof(uint64_t));
hipMemset(d_MAT_IDX55d9d61bd2d0, 0, sizeof(uint64_t));
auto MAT55d9d61bd2d0n1___n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT55d9d61bd2d0);
DBI16Type* d_MAT55d9d61bd2d0n1___n_name_encoded;
hipMalloc(&d_MAT55d9d61bd2d0n1___n_name_encoded, sizeof(DBI16Type) * COUNT55d9d61bd2d0);
auto MAT55d9d61bd2d0n2___n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT55d9d61bd2d0);
DBI16Type* d_MAT55d9d61bd2d0n2___n_name_encoded;
hipMalloc(&d_MAT55d9d61bd2d0n2___n_name_encoded, sizeof(DBI16Type) * COUNT55d9d61bd2d0);
auto MAT55d9d61bd2d0map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT55d9d61bd2d0);
DBI64Type* d_MAT55d9d61bd2d0map0__tmp_attr0;
hipMalloc(&d_MAT55d9d61bd2d0map0__tmp_attr0, sizeof(DBI64Type) * COUNT55d9d61bd2d0);
auto MAT55d9d61bd2d0aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT55d9d61bd2d0);
DBDecimalType* d_MAT55d9d61bd2d0aggr0__tmp_attr2;
hipMalloc(&d_MAT55d9d61bd2d0aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT55d9d61bd2d0);
main_55d9d6217d40<<<std::ceil((float)COUNT55d9d61a8de0/32.), 32>>>(COUNT55d9d61a8de0, d_MAT55d9d61bd2d0aggr0__tmp_attr2, d_MAT55d9d61bd2d0map0__tmp_attr0, d_MAT55d9d61bd2d0n1___n_name_encoded, d_MAT55d9d61bd2d0n2___n_name_encoded, d_MAT_IDX55d9d61bd2d0, d_aggr0__tmp_attr2, d_KEY_55d9d61a8de0map0__tmp_attr0, d_KEY_55d9d61a8de0n1___n_name_encoded, d_KEY_55d9d61a8de0n2___n_name_encoded);
hipMemcpy(MAT55d9d61bd2d0n1___n_name_encoded, d_MAT55d9d61bd2d0n1___n_name_encoded, sizeof(DBI16Type) * COUNT55d9d61bd2d0, hipMemcpyDeviceToHost);
hipMemcpy(MAT55d9d61bd2d0n2___n_name_encoded, d_MAT55d9d61bd2d0n2___n_name_encoded, sizeof(DBI16Type) * COUNT55d9d61bd2d0, hipMemcpyDeviceToHost);
hipMemcpy(MAT55d9d61bd2d0map0__tmp_attr0, d_MAT55d9d61bd2d0map0__tmp_attr0, sizeof(DBI64Type) * COUNT55d9d61bd2d0, hipMemcpyDeviceToHost);
hipMemcpy(MAT55d9d61bd2d0aggr0__tmp_attr2, d_MAT55d9d61bd2d0aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT55d9d61bd2d0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT55d9d61bd2d0; i++) { std::cout << n1___n_name_map[MAT55d9d61bd2d0n1___n_name_encoded[i]] << "\t";
std::cout << n2___n_name_map[MAT55d9d61bd2d0n2___n_name_encoded[i]] << "\t";
std::cout << MAT55d9d61bd2d0map0__tmp_attr0[i] << "\t";
std::cout << MAT55d9d61bd2d0aggr0__tmp_attr2[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_55d9d61dc060);
hipFree(d_BUF_IDX_55d9d61dc060);
hipFree(d_COUNT55d9d61dc060);
hipFree(d_BUF_55d9d61fb1f0);
hipFree(d_BUF_IDX_55d9d61fb1f0);
hipFree(d_COUNT55d9d61fb1f0);
hipFree(d_BUF_55d9d61f0350);
hipFree(d_BUF_IDX_55d9d61f0350);
hipFree(d_COUNT55d9d61f0350);
hipFree(d_BUF_55d9d61f2090);
hipFree(d_BUF_IDX_55d9d61f2090);
hipFree(d_COUNT55d9d61f2090);
hipFree(d_BUF_55d9d61f21a0);
hipFree(d_BUF_IDX_55d9d61f21a0);
hipFree(d_COUNT55d9d61f21a0);
hipFree(d_KEY_55d9d61a8de0map0__tmp_attr0);
hipFree(d_KEY_55d9d61a8de0n1___n_name_encoded);
hipFree(d_KEY_55d9d61a8de0n2___n_name_encoded);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_COUNT55d9d61bd2d0);
hipFree(d_MAT55d9d61bd2d0aggr0__tmp_attr2);
hipFree(d_MAT55d9d61bd2d0map0__tmp_attr0);
hipFree(d_MAT55d9d61bd2d0n1___n_name_encoded);
hipFree(d_MAT55d9d61bd2d0n2___n_name_encoded);
hipFree(d_MAT_IDX55d9d61bd2d0);
free(MAT55d9d61bd2d0aggr0__tmp_attr2);
free(MAT55d9d61bd2d0map0__tmp_attr0);
free(MAT55d9d61bd2d0n1___n_name_encoded);
free(MAT55d9d61bd2d0n2___n_name_encoded);
}