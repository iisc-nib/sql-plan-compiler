#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_611b03fad7f0(uint64_t* COUNT611b03fa7e60, DBStringType* n1___n_name, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n1___n_name = n1___n_name[tid];
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
//Materialize count
atomicAdd((int*)COUNT611b03fa7e60, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_611b03fad7f0(uint64_t* BUF_611b03fa7e60, uint64_t* BUF_IDX_611b03fa7e60, HASHTABLE_INSERT HT_611b03fa7e60, DBStringType* n1___n_name, DBI32Type* n1___n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n1___n_name = n1___n_name[tid];
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
uint64_t KEY_611b03fa7e60 = 0;
auto reg_n1___n_nationkey = n1___n_nationkey[tid];

KEY_611b03fa7e60 |= reg_n1___n_nationkey;
// Insert hash table kernel;
auto buf_idx_611b03fa7e60 = atomicAdd((int*)BUF_IDX_611b03fa7e60, 1);
HT_611b03fa7e60.insert(cuco::pair{KEY_611b03fa7e60, buf_idx_611b03fa7e60});
BUF_611b03fa7e60[buf_idx_611b03fa7e60 * 1 + 0] = tid;
}
__global__ void count_611b03fb1360(uint64_t* COUNT611b03fa7b40, DBStringType* n2___n_name, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n2___n_name = n2___n_name[tid];
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
//Materialize count
atomicAdd((int*)COUNT611b03fa7b40, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_611b03fb1360(uint64_t* BUF_611b03fa7b40, uint64_t* BUF_IDX_611b03fa7b40, HASHTABLE_INSERT HT_611b03fa7b40, DBStringType* n2___n_name, DBI32Type* n2___n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n2___n_name = n2___n_name[tid];
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
uint64_t KEY_611b03fa7b40 = 0;
auto reg_n2___n_nationkey = n2___n_nationkey[tid];

KEY_611b03fa7b40 |= reg_n2___n_nationkey;
// Insert hash table kernel;
auto buf_idx_611b03fa7b40 = atomicAdd((int*)BUF_IDX_611b03fa7b40, 1);
HT_611b03fa7b40.insert(cuco::pair{KEY_611b03fa7b40, buf_idx_611b03fa7b40});
BUF_611b03fa7b40[buf_idx_611b03fa7b40 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_611b03fb26f0(uint64_t* BUF_611b03fa7b40, uint64_t* COUNT611b03f9e270, HASHTABLE_PROBE HT_611b03fa7b40, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_611b03fa7b40 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_611b03fa7b40 |= reg_customer__c_nationkey;
//Probe Hash table
HT_611b03fa7b40.for_each(KEY_611b03fa7b40, [&] __device__ (auto const SLOT_611b03fa7b40) {

auto const [slot_first611b03fa7b40, slot_second611b03fa7b40] = SLOT_611b03fa7b40;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT611b03f9e270, 1);
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_611b03fb26f0(uint64_t* BUF_611b03f9e270, uint64_t* BUF_611b03fa7b40, uint64_t* BUF_IDX_611b03f9e270, HASHTABLE_INSERT HT_611b03f9e270, HASHTABLE_PROBE HT_611b03fa7b40, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_611b03fa7b40 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_611b03fa7b40 |= reg_customer__c_nationkey;
//Probe Hash table
HT_611b03fa7b40.for_each(KEY_611b03fa7b40, [&] __device__ (auto const SLOT_611b03fa7b40) {
auto const [slot_first611b03fa7b40, slot_second611b03fa7b40] = SLOT_611b03fa7b40;
if (!(true)) return;
uint64_t KEY_611b03f9e270 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_611b03f9e270 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_611b03f9e270 = atomicAdd((int*)BUF_IDX_611b03f9e270, 1);
HT_611b03f9e270.insert(cuco::pair{KEY_611b03f9e270, buf_idx_611b03f9e270});
BUF_611b03f9e270[buf_idx_611b03f9e270 * 2 + 0] = BUF_611b03fa7b40[slot_second611b03fa7b40 * 1 + 0];
BUF_611b03f9e270[buf_idx_611b03f9e270 * 2 + 1] = tid;
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_611b03fab9c0(uint64_t* BUF_611b03f9e270, uint64_t* COUNT611b03f9d9e0, HASHTABLE_PROBE HT_611b03f9e270, DBI32Type* orders__o_custkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_611b03f9e270 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_611b03f9e270 |= reg_orders__o_custkey;
//Probe Hash table
HT_611b03f9e270.for_each(KEY_611b03f9e270, [&] __device__ (auto const SLOT_611b03f9e270) {

auto const [slot_first611b03f9e270, slot_second611b03f9e270] = SLOT_611b03f9e270;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT611b03f9d9e0, 1);
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_611b03fab9c0(uint64_t* BUF_611b03f9d9e0, uint64_t* BUF_611b03f9e270, uint64_t* BUF_IDX_611b03f9d9e0, HASHTABLE_INSERT HT_611b03f9d9e0, HASHTABLE_PROBE HT_611b03f9e270, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_611b03f9e270 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_611b03f9e270 |= reg_orders__o_custkey;
//Probe Hash table
HT_611b03f9e270.for_each(KEY_611b03f9e270, [&] __device__ (auto const SLOT_611b03f9e270) {
auto const [slot_first611b03f9e270, slot_second611b03f9e270] = SLOT_611b03f9e270;
if (!(true)) return;
uint64_t KEY_611b03f9d9e0 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_611b03f9d9e0 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_611b03f9d9e0 = atomicAdd((int*)BUF_IDX_611b03f9d9e0, 1);
HT_611b03f9d9e0.insert(cuco::pair{KEY_611b03f9d9e0, buf_idx_611b03f9d9e0});
BUF_611b03f9d9e0[buf_idx_611b03f9d9e0 * 3 + 0] = tid;
BUF_611b03f9d9e0[buf_idx_611b03f9d9e0 * 3 + 1] = BUF_611b03f9e270[slot_second611b03f9e270 * 2 + 0];
BUF_611b03f9d9e0[buf_idx_611b03f9d9e0 * 3 + 2] = BUF_611b03f9e270[slot_second611b03f9e270 * 2 + 1];
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_611b03f88730(uint64_t* BUF_611b03fa7e60, uint64_t* COUNT611b03f9ffa0, HASHTABLE_PROBE HT_611b03fa7e60, DBI32Type* supplier__s_nationkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_611b03fa7e60 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_611b03fa7e60 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_611b03fa7e60.for_each(KEY_611b03fa7e60, [&] __device__ (auto const SLOT_611b03fa7e60) {

auto const [slot_first611b03fa7e60, slot_second611b03fa7e60] = SLOT_611b03fa7e60;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT611b03f9ffa0, 1);
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_611b03f88730(uint64_t* BUF_611b03f9ffa0, uint64_t* BUF_611b03fa7e60, uint64_t* BUF_IDX_611b03f9ffa0, HASHTABLE_INSERT HT_611b03f9ffa0, HASHTABLE_PROBE HT_611b03fa7e60, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_611b03fa7e60 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_611b03fa7e60 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_611b03fa7e60.for_each(KEY_611b03fa7e60, [&] __device__ (auto const SLOT_611b03fa7e60) {
auto const [slot_first611b03fa7e60, slot_second611b03fa7e60] = SLOT_611b03fa7e60;
if (!(true)) return;
uint64_t KEY_611b03f9ffa0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_611b03f9ffa0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_611b03f9ffa0 = atomicAdd((int*)BUF_IDX_611b03f9ffa0, 1);
HT_611b03f9ffa0.insert(cuco::pair{KEY_611b03f9ffa0, buf_idx_611b03f9ffa0});
BUF_611b03f9ffa0[buf_idx_611b03f9ffa0 * 2 + 0] = tid;
BUF_611b03f9ffa0[buf_idx_611b03f9ffa0 * 2 + 1] = BUF_611b03fa7e60[slot_second611b03fa7e60 * 1 + 0];
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_611b03f88d00(uint64_t* BUF_611b03f9d9e0, uint64_t* BUF_611b03f9ffa0, HASHTABLE_INSERT HT_611b03f57ca0, HASHTABLE_PROBE HT_611b03f9d9e0, HASHTABLE_PROBE HT_611b03f9ffa0, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n1___n_name, DBI16Type* n1___n_name_encoded, DBStringType* n2___n_name, DBI16Type* n2___n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9861, Predicate::lte))) return;
uint64_t KEY_611b03f9d9e0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_611b03f9d9e0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_611b03f9d9e0.for_each(KEY_611b03f9d9e0, [&] __device__ (auto const SLOT_611b03f9d9e0) {

auto const [slot_first611b03f9d9e0, slot_second611b03f9d9e0] = SLOT_611b03f9d9e0;
if (!(true)) return;
uint64_t KEY_611b03f9ffa0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_611b03f9ffa0 |= reg_lineitem__l_suppkey;
//Probe Hash table
HT_611b03f9ffa0.for_each(KEY_611b03f9ffa0, [&] __device__ (auto const SLOT_611b03f9ffa0) {

auto const [slot_first611b03f9ffa0, slot_second611b03f9ffa0] = SLOT_611b03f9ffa0;
auto reg_n1___n_name = n1___n_name[BUF_611b03f9ffa0[slot_second611b03f9ffa0 * 2 + 1]];
auto reg_n2___n_name = n2___n_name[BUF_611b03f9d9e0[slot_second611b03f9d9e0 * 3 + 1]];
if (!((((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq))) || ((evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) && (true))) return;
uint64_t KEY_611b03f57ca0 = 0;
auto reg_n1___n_name_encoded = n1___n_name_encoded[BUF_611b03f9ffa0[slot_second611b03f9ffa0 * 2 + 1]];

KEY_611b03f57ca0 |= reg_n1___n_name_encoded;
auto reg_n2___n_name_encoded = n2___n_name_encoded[BUF_611b03f9d9e0[slot_second611b03f9d9e0 * 3 + 1]];
KEY_611b03f57ca0 <<= 16;
KEY_611b03f57ca0 |= reg_n2___n_name_encoded;
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_lineitem__l_shipdate);
KEY_611b03f57ca0 <<= 32;
KEY_611b03f57ca0 |= (DBI32Type)reg_map0__tmp_attr0;
//Create aggregation hash table
HT_611b03f57ca0.insert(cuco::pair{KEY_611b03f57ca0, 1});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_611b03f88d00(uint64_t* BUF_611b03f9d9e0, uint64_t* BUF_611b03f9ffa0, HASHTABLE_FIND HT_611b03f57ca0, HASHTABLE_PROBE HT_611b03f9d9e0, HASHTABLE_PROBE HT_611b03f9ffa0, DBI64Type* KEY_611b03f57ca0map0__tmp_attr0, DBI16Type* KEY_611b03f57ca0n1___n_name_encoded, DBI16Type* KEY_611b03f57ca0n2___n_name_encoded, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n1___n_name, DBI16Type* n1___n_name_encoded, DBStringType* n2___n_name, DBI16Type* n2___n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9861, Predicate::lte))) return;
uint64_t KEY_611b03f9d9e0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_611b03f9d9e0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_611b03f9d9e0.for_each(KEY_611b03f9d9e0, [&] __device__ (auto const SLOT_611b03f9d9e0) {
auto const [slot_first611b03f9d9e0, slot_second611b03f9d9e0] = SLOT_611b03f9d9e0;
if (!(true)) return;
uint64_t KEY_611b03f9ffa0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_611b03f9ffa0 |= reg_lineitem__l_suppkey;
//Probe Hash table
HT_611b03f9ffa0.for_each(KEY_611b03f9ffa0, [&] __device__ (auto const SLOT_611b03f9ffa0) {
auto const [slot_first611b03f9ffa0, slot_second611b03f9ffa0] = SLOT_611b03f9ffa0;
auto reg_n1___n_name = n1___n_name[BUF_611b03f9ffa0[slot_second611b03f9ffa0 * 2 + 1]];
auto reg_n2___n_name = n2___n_name[BUF_611b03f9d9e0[slot_second611b03f9d9e0 * 3 + 1]];
if (!((((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq))) || ((evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) && (true))) return;
uint64_t KEY_611b03f57ca0 = 0;
auto reg_n1___n_name_encoded = n1___n_name_encoded[BUF_611b03f9ffa0[slot_second611b03f9ffa0 * 2 + 1]];

KEY_611b03f57ca0 |= reg_n1___n_name_encoded;
auto reg_n2___n_name_encoded = n2___n_name_encoded[BUF_611b03f9d9e0[slot_second611b03f9d9e0 * 3 + 1]];
KEY_611b03f57ca0 <<= 16;
KEY_611b03f57ca0 |= reg_n2___n_name_encoded;
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_lineitem__l_shipdate);
KEY_611b03f57ca0 <<= 32;
KEY_611b03f57ca0 |= (DBI32Type)reg_map0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_611b03f57ca0 = HT_611b03f57ca0.find(KEY_611b03f57ca0)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_611b03f57ca0], reg_map0__tmp_attr1);
KEY_611b03f57ca0n1___n_name_encoded[buf_idx_611b03f57ca0] = reg_n1___n_name_encoded;
KEY_611b03f57ca0n2___n_name_encoded[buf_idx_611b03f57ca0] = reg_n2___n_name_encoded;
KEY_611b03f57ca0map0__tmp_attr0[buf_idx_611b03f57ca0] = reg_map0__tmp_attr0;
});
});
}
__global__ void count_611b03fc6700(size_t COUNT611b03f57ca0, uint64_t* COUNT611b03f6b3a0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT611b03f57ca0) return;
//Materialize count
atomicAdd((int*)COUNT611b03f6b3a0, 1);
}
__global__ void main_611b03fc6700(size_t COUNT611b03f57ca0, DBDecimalType* MAT611b03f6b3a0aggr0__tmp_attr2, DBI64Type* MAT611b03f6b3a0map0__tmp_attr0, DBI16Type* MAT611b03f6b3a0n1___n_name_encoded, DBI16Type* MAT611b03f6b3a0n2___n_name_encoded, uint64_t* MAT_IDX611b03f6b3a0, DBDecimalType* aggr0__tmp_attr2, DBI64Type* map0__tmp_attr0, DBI16Type* n1___n_name_encoded, DBI16Type* n2___n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT611b03f57ca0) return;
//Materialize buffers
auto mat_idx611b03f6b3a0 = atomicAdd((int*)MAT_IDX611b03f6b3a0, 1);
auto reg_n1___n_name_encoded = n1___n_name_encoded[tid];
MAT611b03f6b3a0n1___n_name_encoded[mat_idx611b03f6b3a0] = reg_n1___n_name_encoded;
auto reg_n2___n_name_encoded = n2___n_name_encoded[tid];
MAT611b03f6b3a0n2___n_name_encoded[mat_idx611b03f6b3a0] = reg_n2___n_name_encoded;
auto reg_map0__tmp_attr0 = map0__tmp_attr0[tid];
MAT611b03f6b3a0map0__tmp_attr0[mat_idx611b03f6b3a0] = reg_map0__tmp_attr0;
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT611b03f6b3a0aggr0__tmp_attr2[mat_idx611b03f6b3a0] = reg_aggr0__tmp_attr2;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map) {
//Materialize count
uint64_t* d_COUNT611b03fa7e60;
hipMalloc(&d_COUNT611b03fa7e60, sizeof(uint64_t));
hipMemset(d_COUNT611b03fa7e60, 0, sizeof(uint64_t));
count_611b03fad7f0<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT611b03fa7e60, d_nation__n_name, nation_size);
uint64_t COUNT611b03fa7e60;
hipMemcpy(&COUNT611b03fa7e60, d_COUNT611b03fa7e60, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_611b03fa7e60;
hipMalloc(&d_BUF_IDX_611b03fa7e60, sizeof(uint64_t));
hipMemset(d_BUF_IDX_611b03fa7e60, 0, sizeof(uint64_t));
uint64_t* d_BUF_611b03fa7e60;
hipMalloc(&d_BUF_611b03fa7e60, sizeof(uint64_t) * COUNT611b03fa7e60 * 1);
auto d_HT_611b03fa7e60 = cuco::experimental::static_multimap{ (int)COUNT611b03fa7e60*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_611b03fad7f0<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_611b03fa7e60, d_BUF_IDX_611b03fa7e60, d_HT_611b03fa7e60.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
//Materialize count
uint64_t* d_COUNT611b03fa7b40;
hipMalloc(&d_COUNT611b03fa7b40, sizeof(uint64_t));
hipMemset(d_COUNT611b03fa7b40, 0, sizeof(uint64_t));
count_611b03fb1360<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT611b03fa7b40, d_nation__n_name, nation_size);
uint64_t COUNT611b03fa7b40;
hipMemcpy(&COUNT611b03fa7b40, d_COUNT611b03fa7b40, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_611b03fa7b40;
hipMalloc(&d_BUF_IDX_611b03fa7b40, sizeof(uint64_t));
hipMemset(d_BUF_IDX_611b03fa7b40, 0, sizeof(uint64_t));
uint64_t* d_BUF_611b03fa7b40;
hipMalloc(&d_BUF_611b03fa7b40, sizeof(uint64_t) * COUNT611b03fa7b40 * 1);
auto d_HT_611b03fa7b40 = cuco::experimental::static_multimap{ (int)COUNT611b03fa7b40*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_611b03fb1360<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_611b03fa7b40, d_BUF_IDX_611b03fa7b40, d_HT_611b03fa7b40.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
//Materialize count
uint64_t* d_COUNT611b03f9e270;
hipMalloc(&d_COUNT611b03f9e270, sizeof(uint64_t));
hipMemset(d_COUNT611b03f9e270, 0, sizeof(uint64_t));
count_611b03fb26f0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_611b03fa7b40, d_COUNT611b03f9e270, d_HT_611b03fa7b40.ref(cuco::for_each), d_customer__c_nationkey, customer_size);
uint64_t COUNT611b03f9e270;
hipMemcpy(&COUNT611b03f9e270, d_COUNT611b03f9e270, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_611b03f9e270;
hipMalloc(&d_BUF_IDX_611b03f9e270, sizeof(uint64_t));
hipMemset(d_BUF_IDX_611b03f9e270, 0, sizeof(uint64_t));
uint64_t* d_BUF_611b03f9e270;
hipMalloc(&d_BUF_611b03f9e270, sizeof(uint64_t) * COUNT611b03f9e270 * 2);
auto d_HT_611b03f9e270 = cuco::experimental::static_multimap{ (int)COUNT611b03f9e270*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_611b03fb26f0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_611b03f9e270, d_BUF_611b03fa7b40, d_BUF_IDX_611b03f9e270, d_HT_611b03f9e270.ref(cuco::insert), d_HT_611b03fa7b40.ref(cuco::for_each), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
//Materialize count
uint64_t* d_COUNT611b03f9d9e0;
hipMalloc(&d_COUNT611b03f9d9e0, sizeof(uint64_t));
hipMemset(d_COUNT611b03f9d9e0, 0, sizeof(uint64_t));
count_611b03fab9c0<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_611b03f9e270, d_COUNT611b03f9d9e0, d_HT_611b03f9e270.ref(cuco::for_each), d_orders__o_custkey, orders_size);
uint64_t COUNT611b03f9d9e0;
hipMemcpy(&COUNT611b03f9d9e0, d_COUNT611b03f9d9e0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_611b03f9d9e0;
hipMalloc(&d_BUF_IDX_611b03f9d9e0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_611b03f9d9e0, 0, sizeof(uint64_t));
uint64_t* d_BUF_611b03f9d9e0;
hipMalloc(&d_BUF_611b03f9d9e0, sizeof(uint64_t) * COUNT611b03f9d9e0 * 3);
auto d_HT_611b03f9d9e0 = cuco::experimental::static_multimap{ (int)COUNT611b03f9d9e0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_611b03fab9c0<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_611b03f9d9e0, d_BUF_611b03f9e270, d_BUF_IDX_611b03f9d9e0, d_HT_611b03f9d9e0.ref(cuco::insert), d_HT_611b03f9e270.ref(cuco::for_each), d_orders__o_custkey, d_orders__o_orderkey, orders_size);
//Materialize count
uint64_t* d_COUNT611b03f9ffa0;
hipMalloc(&d_COUNT611b03f9ffa0, sizeof(uint64_t));
hipMemset(d_COUNT611b03f9ffa0, 0, sizeof(uint64_t));
count_611b03f88730<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_611b03fa7e60, d_COUNT611b03f9ffa0, d_HT_611b03fa7e60.ref(cuco::for_each), d_supplier__s_nationkey, supplier_size);
uint64_t COUNT611b03f9ffa0;
hipMemcpy(&COUNT611b03f9ffa0, d_COUNT611b03f9ffa0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_611b03f9ffa0;
hipMalloc(&d_BUF_IDX_611b03f9ffa0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_611b03f9ffa0, 0, sizeof(uint64_t));
uint64_t* d_BUF_611b03f9ffa0;
hipMalloc(&d_BUF_611b03f9ffa0, sizeof(uint64_t) * COUNT611b03f9ffa0 * 2);
auto d_HT_611b03f9ffa0 = cuco::experimental::static_multimap{ (int)COUNT611b03f9ffa0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_611b03f88730<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_611b03f9ffa0, d_BUF_611b03fa7e60, d_BUF_IDX_611b03f9ffa0, d_HT_611b03f9ffa0.ref(cuco::insert), d_HT_611b03fa7e60.ref(cuco::for_each), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
//Create aggregation hash table
auto d_HT_611b03f57ca0 = cuco::static_map{ (int)13634*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_611b03f88d00<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_611b03f9d9e0, d_BUF_611b03f9ffa0, d_HT_611b03f57ca0.ref(cuco::insert), d_HT_611b03f9d9e0.ref(cuco::for_each), d_HT_611b03f9ffa0.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_nation__n_name_encoded, d_nation__n_name, d_nation__n_name_encoded);
size_t COUNT611b03f57ca0 = d_HT_611b03f57ca0.size();
thrust::device_vector<int64_t> keys_611b03f57ca0(COUNT611b03f57ca0), vals_611b03f57ca0(COUNT611b03f57ca0);
d_HT_611b03f57ca0.retrieve_all(keys_611b03f57ca0.begin(), vals_611b03f57ca0.begin());
d_HT_611b03f57ca0.clear();
int64_t* raw_keys611b03f57ca0 = thrust::raw_pointer_cast(keys_611b03f57ca0.data());
insertKeys<<<std::ceil((float)COUNT611b03f57ca0/32.), 32>>>(raw_keys611b03f57ca0, d_HT_611b03f57ca0.ref(cuco::insert), COUNT611b03f57ca0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT611b03f57ca0);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT611b03f57ca0);
DBI16Type* d_KEY_611b03f57ca0n1___n_name_encoded;
hipMalloc(&d_KEY_611b03f57ca0n1___n_name_encoded, sizeof(DBI16Type) * COUNT611b03f57ca0);
hipMemset(d_KEY_611b03f57ca0n1___n_name_encoded, 0, sizeof(DBI16Type) * COUNT611b03f57ca0);
DBI16Type* d_KEY_611b03f57ca0n2___n_name_encoded;
hipMalloc(&d_KEY_611b03f57ca0n2___n_name_encoded, sizeof(DBI16Type) * COUNT611b03f57ca0);
hipMemset(d_KEY_611b03f57ca0n2___n_name_encoded, 0, sizeof(DBI16Type) * COUNT611b03f57ca0);
DBI64Type* d_KEY_611b03f57ca0map0__tmp_attr0;
hipMalloc(&d_KEY_611b03f57ca0map0__tmp_attr0, sizeof(DBI64Type) * COUNT611b03f57ca0);
hipMemset(d_KEY_611b03f57ca0map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT611b03f57ca0);
main_611b03f88d00<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_611b03f9d9e0, d_BUF_611b03f9ffa0, d_HT_611b03f57ca0.ref(cuco::find), d_HT_611b03f9d9e0.ref(cuco::for_each), d_HT_611b03f9ffa0.ref(cuco::for_each), d_KEY_611b03f57ca0map0__tmp_attr0, d_KEY_611b03f57ca0n1___n_name_encoded, d_KEY_611b03f57ca0n2___n_name_encoded, d_aggr0__tmp_attr2, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_nation__n_name_encoded, d_nation__n_name, d_nation__n_name_encoded);
//Materialize count
uint64_t* d_COUNT611b03f6b3a0;
hipMalloc(&d_COUNT611b03f6b3a0, sizeof(uint64_t));
hipMemset(d_COUNT611b03f6b3a0, 0, sizeof(uint64_t));
count_611b03fc6700<<<std::ceil((float)COUNT611b03f57ca0/32.), 32>>>(COUNT611b03f57ca0, d_COUNT611b03f6b3a0);
uint64_t COUNT611b03f6b3a0;
hipMemcpy(&COUNT611b03f6b3a0, d_COUNT611b03f6b3a0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX611b03f6b3a0;
hipMalloc(&d_MAT_IDX611b03f6b3a0, sizeof(uint64_t));
hipMemset(d_MAT_IDX611b03f6b3a0, 0, sizeof(uint64_t));
auto MAT611b03f6b3a0n1___n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT611b03f6b3a0);
DBI16Type* d_MAT611b03f6b3a0n1___n_name_encoded;
hipMalloc(&d_MAT611b03f6b3a0n1___n_name_encoded, sizeof(DBI16Type) * COUNT611b03f6b3a0);
auto MAT611b03f6b3a0n2___n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT611b03f6b3a0);
DBI16Type* d_MAT611b03f6b3a0n2___n_name_encoded;
hipMalloc(&d_MAT611b03f6b3a0n2___n_name_encoded, sizeof(DBI16Type) * COUNT611b03f6b3a0);
auto MAT611b03f6b3a0map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT611b03f6b3a0);
DBI64Type* d_MAT611b03f6b3a0map0__tmp_attr0;
hipMalloc(&d_MAT611b03f6b3a0map0__tmp_attr0, sizeof(DBI64Type) * COUNT611b03f6b3a0);
auto MAT611b03f6b3a0aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT611b03f6b3a0);
DBDecimalType* d_MAT611b03f6b3a0aggr0__tmp_attr2;
hipMalloc(&d_MAT611b03f6b3a0aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT611b03f6b3a0);
main_611b03fc6700<<<std::ceil((float)COUNT611b03f57ca0/32.), 32>>>(COUNT611b03f57ca0, d_MAT611b03f6b3a0aggr0__tmp_attr2, d_MAT611b03f6b3a0map0__tmp_attr0, d_MAT611b03f6b3a0n1___n_name_encoded, d_MAT611b03f6b3a0n2___n_name_encoded, d_MAT_IDX611b03f6b3a0, d_aggr0__tmp_attr2, d_KEY_611b03f57ca0map0__tmp_attr0, d_KEY_611b03f57ca0n1___n_name_encoded, d_KEY_611b03f57ca0n2___n_name_encoded);
hipMemcpy(MAT611b03f6b3a0n1___n_name_encoded, d_MAT611b03f6b3a0n1___n_name_encoded, sizeof(DBI16Type) * COUNT611b03f6b3a0, hipMemcpyDeviceToHost);
hipMemcpy(MAT611b03f6b3a0n2___n_name_encoded, d_MAT611b03f6b3a0n2___n_name_encoded, sizeof(DBI16Type) * COUNT611b03f6b3a0, hipMemcpyDeviceToHost);
hipMemcpy(MAT611b03f6b3a0map0__tmp_attr0, d_MAT611b03f6b3a0map0__tmp_attr0, sizeof(DBI64Type) * COUNT611b03f6b3a0, hipMemcpyDeviceToHost);
hipMemcpy(MAT611b03f6b3a0aggr0__tmp_attr2, d_MAT611b03f6b3a0aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT611b03f6b3a0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT611b03f6b3a0; i++) { std::cout << n1___n_name_map[MAT611b03f6b3a0n1___n_name_encoded[i]] << "\t";
std::cout << n2___n_name_map[MAT611b03f6b3a0n2___n_name_encoded[i]] << "\t";
std::cout << MAT611b03f6b3a0map0__tmp_attr0[i] << "\t";
std::cout << MAT611b03f6b3a0aggr0__tmp_attr2[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_611b03fa7e60);
hipFree(d_BUF_IDX_611b03fa7e60);
hipFree(d_COUNT611b03fa7e60);
hipFree(d_BUF_611b03fa7b40);
hipFree(d_BUF_IDX_611b03fa7b40);
hipFree(d_COUNT611b03fa7b40);
hipFree(d_BUF_611b03f9e270);
hipFree(d_BUF_IDX_611b03f9e270);
hipFree(d_COUNT611b03f9e270);
hipFree(d_BUF_611b03f9d9e0);
hipFree(d_BUF_IDX_611b03f9d9e0);
hipFree(d_COUNT611b03f9d9e0);
hipFree(d_BUF_611b03f9ffa0);
hipFree(d_BUF_IDX_611b03f9ffa0);
hipFree(d_COUNT611b03f9ffa0);
hipFree(d_KEY_611b03f57ca0map0__tmp_attr0);
hipFree(d_KEY_611b03f57ca0n1___n_name_encoded);
hipFree(d_KEY_611b03f57ca0n2___n_name_encoded);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_COUNT611b03f6b3a0);
hipFree(d_MAT611b03f6b3a0aggr0__tmp_attr2);
hipFree(d_MAT611b03f6b3a0map0__tmp_attr0);
hipFree(d_MAT611b03f6b3a0n1___n_name_encoded);
hipFree(d_MAT611b03f6b3a0n2___n_name_encoded);
hipFree(d_MAT_IDX611b03f6b3a0);
free(MAT611b03f6b3a0aggr0__tmp_attr2);
free(MAT611b03f6b3a0map0__tmp_attr0);
free(MAT611b03f6b3a0n1___n_name_encoded);
free(MAT611b03f6b3a0n2___n_name_encoded);
}