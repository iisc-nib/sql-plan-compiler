#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_INSERT HT_4, DBI32Type* customer__c_custkey, DBStringType* customer__c_mktsegment, size_t customer_size, int64_t* cycles_per_warp_main_1_join_build_4, int64_t* cycles_per_warp_main_1_selection_0) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_customer__c_mktsegment[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_mktsegment[ITEM] = customer__c_mktsegment[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_customer__c_mktsegment[ITEM], "BUILDING", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_4.insert(cuco::pair{KEY_4[ITEM], ITEM*TB + tid});
BUF_4[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_build_4[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_4, uint64_t* BUF_7, uint64_t* BUF_IDX_7, HASHTABLE_PROBE HT_4, HASHTABLE_INSERT HT_7, int64_t* cycles_per_warp_main_3_join_build_7, int64_t* cycles_per_warp_main_3_join_probe_4, int64_t* cycles_per_warp_main_3_selection_2, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_orders__o_orderdate[ITEM], 9204, Predicate::lt);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_custkey[ITEM] = orders__o_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_orders__o_custkey[ITEM];
}
//Probe Hash table
int64_t slot_second4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0; continue;}
slot_second4[ITEM] = SLOT_4->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_probe_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_7[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_7[ITEM] = 0;
KEY_7[ITEM] |= reg_orders__o_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_7.insert(cuco::pair{KEY_7[ITEM], ITEM*TB + tid});
BUF_7[(ITEM*TB + tid) * 2 + 0] = BUF_4[slot_second4[ITEM] * 1 + 0];
BUF_7[(ITEM*TB + tid) * 2 + 1] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_build_7[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_6(uint64_t* BUF_7, HASHTABLE_PROBE HT_7, HASHTABLE_FIND HT_9, DBI32Type* KEY_9lineitem__l_orderkey, int* SLOT_COUNT_9, DBDecimalType* aggr0__tmp_attr0, DBDateType* aggr__o_orderdate, DBI32Type* aggr__o_shippriority, int64_t* cycles_per_warp_main_6_aggregation_9, int64_t* cycles_per_warp_main_6_join_probe_7, int64_t* cycles_per_warp_main_6_map_8, int64_t* cycles_per_warp_main_6_selection_5, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, size_t lineitem_size, DBDateType* orders__o_orderdate, DBI32Type* orders__o_shippriority) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBDateType reg_lineitem__l_shipdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipdate[ITEM] = lineitem__l_shipdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 9204, Predicate::gt);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_6_selection_5[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_7[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_7[ITEM] = 0;
KEY_7[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
//Probe Hash table
int64_t slot_second7[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_7 = HT_7.find(KEY_7[ITEM]);
if (SLOT_7 == HT_7.end()) {selection_flags[ITEM] = 0; continue;}
slot_second7[ITEM] = SLOT_7->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_6_join_probe_7[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_6_map_8[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_9[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_9[ITEM] = 0;
KEY_9[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]));
}
DBI32Type reg_orders__o_shippriority[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_shippriority[ITEM] = orders__o_shippriority[BUF_7[slot_second7[ITEM] * 2 + 1]];
}
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[BUF_7[slot_second7[ITEM] * 2 + 1]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_9 = get_aggregation_slot(KEY_9[ITEM], HT_9, SLOT_COUNT_9);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_9], reg_map0__tmp_attr1[ITEM]);
aggregate_any(&aggr__o_shippriority[buf_idx_9], reg_orders__o_shippriority[ITEM]);
aggregate_any(&aggr__o_orderdate[buf_idx_9], reg_orders__o_orderdate[ITEM]);
KEY_9lineitem__l_orderkey[buf_idx_9] = reg_lineitem__l_orderkey[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_6_aggregation_9[blockIdx.x] = cycles_per_warp;}
}
__global__ void main_11(size_t COUNT9, DBDecimalType* MAT10aggr0__tmp_attr0, DBDateType* MAT10aggr__o_orderdate, DBI32Type* MAT10aggr__o_shippriority, DBI32Type* MAT10lineitem__l_orderkey, uint64_t* MAT_IDX10, DBDecimalType* aggr0__tmp_attr0, DBDateType* aggr__o_orderdate, DBI32Type* aggr__o_shippriority, int64_t* cycles_per_warp_main_11_materialize_10, DBI32Type* lineitem__l_orderkey) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
DBDateType reg_aggr__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
reg_aggr__o_orderdate[ITEM] = aggr__o_orderdate[ITEM*TB + tid];
}
DBI32Type reg_aggr__o_shippriority[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
reg_aggr__o_shippriority[ITEM] = aggr__o_shippriority[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx10 = atomicAdd((int*)MAT_IDX10, 1);
MAT10lineitem__l_orderkey[mat_idx10] = reg_lineitem__l_orderkey[ITEM];
MAT10aggr0__tmp_attr0[mat_idx10] = reg_aggr0__tmp_attr0[ITEM];
MAT10aggr__o_orderdate[mat_idx10] = reg_aggr__o_orderdate[ITEM];
MAT10aggr__o_shippriority[mat_idx10] = reg_aggr__o_shippriority[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_materialize_10[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_join_build_4;
auto main_1_join_build_4_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_build_4, sizeof(int64_t) * main_1_join_build_4_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_build_4, -1, sizeof(int64_t) * main_1_join_build_4_cpw_size);
size_t COUNT4 = customer_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 1);
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)customer_size/(float)TILE_SIZE), TB>>>(d_BUF_4, d_BUF_IDX_4, d_HT_4.ref(cuco::insert), d_customer__c_custkey, d_customer__c_mktsegment, customer_size, d_cycles_per_warp_main_1_join_build_4, d_cycles_per_warp_main_1_selection_0);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_build_4 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_build_4_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_build_4, d_cycles_per_warp_main_1_join_build_4, sizeof(int64_t) * main_1_join_build_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_build_4 ";
for (auto i=0ull; i < main_1_join_build_4_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_build_4[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_3_selection_2;
auto main_3_selection_2_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_selection_2, sizeof(int64_t) * main_3_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_3_selection_2, -1, sizeof(int64_t) * main_3_selection_2_cpw_size);
int64_t* d_cycles_per_warp_main_3_join_probe_4;
auto main_3_join_probe_4_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_probe_4, sizeof(int64_t) * main_3_join_probe_4_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_probe_4, -1, sizeof(int64_t) * main_3_join_probe_4_cpw_size);
int64_t* d_cycles_per_warp_main_3_join_build_7;
auto main_3_join_build_7_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_build_7, sizeof(int64_t) * main_3_join_build_7_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_build_7, -1, sizeof(int64_t) * main_3_join_build_7_cpw_size);
size_t COUNT7 = orders_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_7;
hipMalloc(&d_BUF_IDX_7, sizeof(uint64_t));
hipMemset(d_BUF_IDX_7, 0, sizeof(uint64_t));
uint64_t* d_BUF_7;
hipMalloc(&d_BUF_7, sizeof(uint64_t) * COUNT7 * 2);
auto d_HT_7 = cuco::static_map{ (int)COUNT7*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)orders_size/(float)TILE_SIZE), TB>>>(d_BUF_4, d_BUF_7, d_BUF_IDX_7, d_HT_4.ref(cuco::find), d_HT_7.ref(cuco::insert), d_cycles_per_warp_main_3_join_build_7, d_cycles_per_warp_main_3_join_probe_4, d_cycles_per_warp_main_3_selection_2, d_orders__o_custkey, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
int64_t* cycles_per_warp_main_3_selection_2 = (int64_t*)malloc(sizeof(int64_t) * main_3_selection_2_cpw_size);
hipMemcpy(cycles_per_warp_main_3_selection_2, d_cycles_per_warp_main_3_selection_2, sizeof(int64_t) * main_3_selection_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_selection_2 ";
for (auto i=0ull; i < main_3_selection_2_cpw_size; i++) std::cout << cycles_per_warp_main_3_selection_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_join_probe_4 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_probe_4_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_probe_4, d_cycles_per_warp_main_3_join_probe_4, sizeof(int64_t) * main_3_join_probe_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_probe_4 ";
for (auto i=0ull; i < main_3_join_probe_4_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_probe_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_join_build_7 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_build_7_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_build_7, d_cycles_per_warp_main_3_join_build_7, sizeof(int64_t) * main_3_join_build_7_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_build_7 ";
for (auto i=0ull; i < main_3_join_build_7_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_build_7[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_6_selection_5;
auto main_6_selection_5_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_6_selection_5, sizeof(int64_t) * main_6_selection_5_cpw_size);
hipMemset(d_cycles_per_warp_main_6_selection_5, -1, sizeof(int64_t) * main_6_selection_5_cpw_size);
int64_t* d_cycles_per_warp_main_6_join_probe_7;
auto main_6_join_probe_7_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_6_join_probe_7, sizeof(int64_t) * main_6_join_probe_7_cpw_size);
hipMemset(d_cycles_per_warp_main_6_join_probe_7, -1, sizeof(int64_t) * main_6_join_probe_7_cpw_size);
int64_t* d_cycles_per_warp_main_6_map_8;
auto main_6_map_8_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_6_map_8, sizeof(int64_t) * main_6_map_8_cpw_size);
hipMemset(d_cycles_per_warp_main_6_map_8, -1, sizeof(int64_t) * main_6_map_8_cpw_size);
int64_t* d_cycles_per_warp_main_6_aggregation_9;
auto main_6_aggregation_9_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_6_aggregation_9, sizeof(int64_t) * main_6_aggregation_9_cpw_size);
hipMemset(d_cycles_per_warp_main_6_aggregation_9, -1, sizeof(int64_t) * main_6_aggregation_9_cpw_size);
size_t COUNT9 = 355555;
auto d_HT_9 = cuco::static_map{ (int)355555*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_9;
hipMalloc(&d_SLOT_COUNT_9, sizeof(int));
hipMemset(d_SLOT_COUNT_9, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT9);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT9);
DBI32Type* d_aggr__o_shippriority;
hipMalloc(&d_aggr__o_shippriority, sizeof(DBI32Type) * COUNT9);
hipMemset(d_aggr__o_shippriority, 0, sizeof(DBI32Type) * COUNT9);
DBDateType* d_aggr__o_orderdate;
hipMalloc(&d_aggr__o_orderdate, sizeof(DBDateType) * COUNT9);
hipMemset(d_aggr__o_orderdate, 0, sizeof(DBDateType) * COUNT9);
DBI32Type* d_KEY_9lineitem__l_orderkey;
hipMalloc(&d_KEY_9lineitem__l_orderkey, sizeof(DBI32Type) * COUNT9);
hipMemset(d_KEY_9lineitem__l_orderkey, 0, sizeof(DBI32Type) * COUNT9);
main_6<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TB>>>(d_BUF_7, d_HT_7.ref(cuco::find), d_HT_9.ref(cuco::insert_and_find), d_KEY_9lineitem__l_orderkey, d_SLOT_COUNT_9, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_cycles_per_warp_main_6_aggregation_9, d_cycles_per_warp_main_6_join_probe_7, d_cycles_per_warp_main_6_map_8, d_cycles_per_warp_main_6_selection_5, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, lineitem_size, d_orders__o_orderdate, d_orders__o_shippriority);
int64_t* cycles_per_warp_main_6_selection_5 = (int64_t*)malloc(sizeof(int64_t) * main_6_selection_5_cpw_size);
hipMemcpy(cycles_per_warp_main_6_selection_5, d_cycles_per_warp_main_6_selection_5, sizeof(int64_t) * main_6_selection_5_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_6_selection_5 ";
for (auto i=0ull; i < main_6_selection_5_cpw_size; i++) std::cout << cycles_per_warp_main_6_selection_5[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_6_join_probe_7 = (int64_t*)malloc(sizeof(int64_t) * main_6_join_probe_7_cpw_size);
hipMemcpy(cycles_per_warp_main_6_join_probe_7, d_cycles_per_warp_main_6_join_probe_7, sizeof(int64_t) * main_6_join_probe_7_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_6_join_probe_7 ";
for (auto i=0ull; i < main_6_join_probe_7_cpw_size; i++) std::cout << cycles_per_warp_main_6_join_probe_7[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_6_map_8 = (int64_t*)malloc(sizeof(int64_t) * main_6_map_8_cpw_size);
hipMemcpy(cycles_per_warp_main_6_map_8, d_cycles_per_warp_main_6_map_8, sizeof(int64_t) * main_6_map_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_6_map_8 ";
for (auto i=0ull; i < main_6_map_8_cpw_size; i++) std::cout << cycles_per_warp_main_6_map_8[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_6_aggregation_9 = (int64_t*)malloc(sizeof(int64_t) * main_6_aggregation_9_cpw_size);
hipMemcpy(cycles_per_warp_main_6_aggregation_9, d_cycles_per_warp_main_6_aggregation_9, sizeof(int64_t) * main_6_aggregation_9_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_6_aggregation_9 ";
for (auto i=0ull; i < main_6_aggregation_9_cpw_size; i++) std::cout << cycles_per_warp_main_6_aggregation_9[i] << " ";
std::cout << std::endl;
COUNT9 = d_HT_9.size();
int64_t* d_cycles_per_warp_main_11_materialize_10;
auto main_11_materialize_10_cpw_size = std::ceil((float)COUNT9/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_materialize_10, sizeof(int64_t) * main_11_materialize_10_cpw_size);
hipMemset(d_cycles_per_warp_main_11_materialize_10, -1, sizeof(int64_t) * main_11_materialize_10_cpw_size);
size_t COUNT10 = COUNT9;
//Materialize buffers
uint64_t* d_MAT_IDX10;
hipMalloc(&d_MAT_IDX10, sizeof(uint64_t));
hipMemset(d_MAT_IDX10, 0, sizeof(uint64_t));
auto MAT10lineitem__l_orderkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT10);
DBI32Type* d_MAT10lineitem__l_orderkey;
hipMalloc(&d_MAT10lineitem__l_orderkey, sizeof(DBI32Type) * COUNT10);
auto MAT10aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT10);
DBDecimalType* d_MAT10aggr0__tmp_attr0;
hipMalloc(&d_MAT10aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT10);
auto MAT10aggr__o_orderdate = (DBDateType*)malloc(sizeof(DBDateType) * COUNT10);
DBDateType* d_MAT10aggr__o_orderdate;
hipMalloc(&d_MAT10aggr__o_orderdate, sizeof(DBDateType) * COUNT10);
auto MAT10aggr__o_shippriority = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT10);
DBI32Type* d_MAT10aggr__o_shippriority;
hipMalloc(&d_MAT10aggr__o_shippriority, sizeof(DBI32Type) * COUNT10);
main_11<<<std::ceil((float)COUNT9/(float)TILE_SIZE), TB>>>(COUNT9, d_MAT10aggr0__tmp_attr0, d_MAT10aggr__o_orderdate, d_MAT10aggr__o_shippriority, d_MAT10lineitem__l_orderkey, d_MAT_IDX10, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_cycles_per_warp_main_11_materialize_10, d_KEY_9lineitem__l_orderkey);
uint64_t MATCOUNT_10 = 0;
hipMemcpy(&MATCOUNT_10, d_MAT_IDX10, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT10lineitem__l_orderkey, d_MAT10lineitem__l_orderkey, sizeof(DBI32Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10aggr0__tmp_attr0, d_MAT10aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10aggr__o_orderdate, d_MAT10aggr__o_orderdate, sizeof(DBDateType) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10aggr__o_shippriority, d_MAT10aggr__o_shippriority, sizeof(DBI32Type) * COUNT10, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_11_materialize_10 = (int64_t*)malloc(sizeof(int64_t) * main_11_materialize_10_cpw_size);
hipMemcpy(cycles_per_warp_main_11_materialize_10, d_cycles_per_warp_main_11_materialize_10, sizeof(int64_t) * main_11_materialize_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_materialize_10 ";
for (auto i=0ull; i < main_11_materialize_10_cpw_size; i++) std::cout << cycles_per_warp_main_11_materialize_10[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_BUF_7);
hipFree(d_BUF_IDX_7);
hipFree(d_KEY_9lineitem__l_orderkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr__o_orderdate);
hipFree(d_aggr__o_shippriority);
hipFree(d_MAT10aggr0__tmp_attr0);
hipFree(d_MAT10aggr__o_orderdate);
hipFree(d_MAT10aggr__o_shippriority);
hipFree(d_MAT10lineitem__l_orderkey);
hipFree(d_MAT_IDX10);
free(MAT10aggr0__tmp_attr0);
free(MAT10aggr__o_orderdate);
free(MAT10aggr__o_shippriority);
free(MAT10lineitem__l_orderkey);
}