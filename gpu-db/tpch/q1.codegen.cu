#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
template<typename HASHTABLE_INSERT>
__global__ void count_1(HASHTABLE_INSERT HT_0, DBCharType* lineitem__l_linestatus, DBCharType* lineitem__l_returnflag, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 10471, Predicate::lte))) return;
uint64_t KEY_0 = 0;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];

KEY_0 |= reg_lineitem__l_returnflag;
auto reg_lineitem__l_linestatus = lineitem__l_linestatus[tid];
KEY_0 <<= 8;
KEY_0 |= reg_lineitem__l_linestatus;
//Create aggregation hash table
HT_0.insert(cuco::pair{KEY_0, 1});
}
template<typename HASHTABLE_FIND>
__global__ void main_1(HASHTABLE_FIND HT_0, DBCharType* KEY_0lineitem__l_linestatus, DBCharType* KEY_0lineitem__l_returnflag, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr0__tmp_attr1, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, DBI64Type* aggr0__tmp_attr9, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, DBDecimalType* aggr_rw__rw2, DBI64Type* aggr_rw__rw3, DBDecimalType* aggr_rw__rw4, DBI64Type* aggr_rw__rw5, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBCharType* lineitem__l_linestatus, DBDecimalType* lineitem__l_quantity, DBCharType* lineitem__l_returnflag, DBDateType* lineitem__l_shipdate, DBDecimalType* lineitem__l_tax, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 10471, Predicate::lte))) return;
uint64_t KEY_0 = 0;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];

KEY_0 |= reg_lineitem__l_returnflag;
auto reg_lineitem__l_linestatus = lineitem__l_linestatus[tid];
KEY_0 <<= 8;
KEY_0 |= reg_lineitem__l_linestatus;
//Aggregate in hashtable
auto buf_idx_0 = HT_0.find(KEY_0)->second;
aggregate_sum(&aggr0__tmp_attr9[buf_idx_0], 1);
auto reg_lineitem__l_tax = lineitem__l_tax[tid];
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr5 = ((reg_lineitem__l_extendedprice) * ((1.0) - (reg_lineitem__l_discount))) * ((1.0) + (reg_lineitem__l_tax));
aggregate_sum(&aggr0__tmp_attr4[buf_idx_0], reg_map0__tmp_attr5);
auto reg_map0__tmp_attr3 = (reg_lineitem__l_extendedprice) * ((1.0) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_0], reg_map0__tmp_attr3);
aggregate_sum(&aggr0__tmp_attr1[buf_idx_0], reg_lineitem__l_extendedprice);
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_0], reg_lineitem__l_quantity);
aggregate_sum(&aggr_rw__rw0[buf_idx_0], reg_lineitem__l_discount);
aggregate_sum(&aggr_rw__rw1[buf_idx_0], 1);
aggregate_sum(&aggr_rw__rw2[buf_idx_0], reg_lineitem__l_extendedprice);
aggregate_sum(&aggr_rw__rw3[buf_idx_0], 1);
aggregate_sum(&aggr_rw__rw4[buf_idx_0], reg_lineitem__l_quantity);
aggregate_sum(&aggr_rw__rw5[buf_idx_0], 1);
KEY_0lineitem__l_returnflag[buf_idx_0] = reg_lineitem__l_returnflag;
KEY_0lineitem__l_linestatus[buf_idx_0] = reg_lineitem__l_linestatus;
}
__global__ void count_3(size_t COUNT0, uint64_t* COUNT2) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT0) return;
//Materialize count
atomicAdd((int*)COUNT2, 1);
}
__global__ void main_3(size_t COUNT0, DBDecimalType* MAT2aggr0__tmp_attr0, DBDecimalType* MAT2aggr0__tmp_attr1, DBDecimalType* MAT2aggr0__tmp_attr2, DBDecimalType* MAT2aggr0__tmp_attr4, DBDecimalType* MAT2aggr0__tmp_attr6, DBDecimalType* MAT2aggr0__tmp_attr7, DBDecimalType* MAT2aggr0__tmp_attr8, DBI64Type* MAT2aggr0__tmp_attr9, DBCharType* MAT2lineitem__l_linestatus, DBCharType* MAT2lineitem__l_returnflag, uint64_t* MAT_IDX2, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr0__tmp_attr1, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, DBI64Type* aggr0__tmp_attr9, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, DBDecimalType* aggr_rw__rw2, DBI64Type* aggr_rw__rw3, DBDecimalType* aggr_rw__rw4, DBI64Type* aggr_rw__rw5, DBCharType* lineitem__l_linestatus, DBCharType* lineitem__l_returnflag) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT0) return;
//Materialize buffers
auto mat_idx2 = atomicAdd((int*)MAT_IDX2, 1);
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];
MAT2lineitem__l_returnflag[mat_idx2] = reg_lineitem__l_returnflag;
auto reg_lineitem__l_linestatus = lineitem__l_linestatus[tid];
MAT2lineitem__l_linestatus[mat_idx2] = reg_lineitem__l_linestatus;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT2aggr0__tmp_attr0[mat_idx2] = reg_aggr0__tmp_attr0;
auto reg_aggr0__tmp_attr1 = aggr0__tmp_attr1[tid];
MAT2aggr0__tmp_attr1[mat_idx2] = reg_aggr0__tmp_attr1;
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT2aggr0__tmp_attr2[mat_idx2] = reg_aggr0__tmp_attr2;
auto reg_aggr0__tmp_attr4 = aggr0__tmp_attr4[tid];
MAT2aggr0__tmp_attr4[mat_idx2] = reg_aggr0__tmp_attr4;
auto reg_aggr_rw__rw5 = aggr_rw__rw5[tid];
auto reg_aggr_rw__rw4 = aggr_rw__rw4[tid];
auto reg_aggr0__tmp_attr6 = (reg_aggr_rw__rw4) / ((DBDecimalType)(reg_aggr_rw__rw5));
MAT2aggr0__tmp_attr6[mat_idx2] = reg_aggr0__tmp_attr6;
auto reg_aggr_rw__rw3 = aggr_rw__rw3[tid];
auto reg_aggr_rw__rw2 = aggr_rw__rw2[tid];
auto reg_aggr0__tmp_attr7 = (reg_aggr_rw__rw2) / ((DBDecimalType)(reg_aggr_rw__rw3));
MAT2aggr0__tmp_attr7[mat_idx2] = reg_aggr0__tmp_attr7;
auto reg_aggr_rw__rw1 = aggr_rw__rw1[tid];
auto reg_aggr_rw__rw0 = aggr_rw__rw0[tid];
auto reg_aggr0__tmp_attr8 = (reg_aggr_rw__rw0) / ((DBDecimalType)(reg_aggr_rw__rw1));
MAT2aggr0__tmp_attr8[mat_idx2] = reg_aggr0__tmp_attr8;
auto reg_aggr0__tmp_attr9 = aggr0__tmp_attr9[tid];
MAT2aggr0__tmp_attr9[mat_idx2] = reg_aggr0__tmp_attr9;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto start = std::chrono::high_resolution_clock::now();
//Create aggregation hash table
auto d_HT_0 = cuco::static_map{ (int)5930889*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_1<<<std::ceil((float)lineitem_size/128.), 128>>>(d_HT_0.ref(cuco::insert), d_lineitem__l_linestatus, d_lineitem__l_returnflag, d_lineitem__l_shipdate, lineitem_size);
size_t COUNT0 = d_HT_0.size();
thrust::device_vector<int64_t> keys_0(COUNT0), vals_0(COUNT0);
d_HT_0.retrieve_all(keys_0.begin(), vals_0.begin());
d_HT_0.clear();
int64_t* raw_keys0 = thrust::raw_pointer_cast(keys_0.data());
insertKeys<<<std::ceil((float)COUNT0/128.), 128>>>(raw_keys0, d_HT_0.ref(cuco::insert), COUNT0);
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr9;
hipMalloc(&d_aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT0);
hipMemset(d_aggr0__tmp_attr9, 0, sizeof(DBI64Type) * COUNT0);
DBDecimalType* d_aggr0__tmp_attr4;
hipMalloc(&d_aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr0__tmp_attr4, 0, sizeof(DBDecimalType) * COUNT0);
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT0);
DBDecimalType* d_aggr0__tmp_attr1;
hipMalloc(&d_aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr0__tmp_attr1, 0, sizeof(DBDecimalType) * COUNT0);
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT0);
DBDecimalType* d_aggr_rw__rw0;
hipMalloc(&d_aggr_rw__rw0, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr_rw__rw0, 0, sizeof(DBDecimalType) * COUNT0);
DBI64Type* d_aggr_rw__rw1;
hipMalloc(&d_aggr_rw__rw1, sizeof(DBI64Type) * COUNT0);
hipMemset(d_aggr_rw__rw1, 0, sizeof(DBI64Type) * COUNT0);
DBDecimalType* d_aggr_rw__rw2;
hipMalloc(&d_aggr_rw__rw2, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr_rw__rw2, 0, sizeof(DBDecimalType) * COUNT0);
DBI64Type* d_aggr_rw__rw3;
hipMalloc(&d_aggr_rw__rw3, sizeof(DBI64Type) * COUNT0);
hipMemset(d_aggr_rw__rw3, 0, sizeof(DBI64Type) * COUNT0);
DBDecimalType* d_aggr_rw__rw4;
hipMalloc(&d_aggr_rw__rw4, sizeof(DBDecimalType) * COUNT0);
hipMemset(d_aggr_rw__rw4, 0, sizeof(DBDecimalType) * COUNT0);
DBI64Type* d_aggr_rw__rw5;
hipMalloc(&d_aggr_rw__rw5, sizeof(DBI64Type) * COUNT0);
hipMemset(d_aggr_rw__rw5, 0, sizeof(DBI64Type) * COUNT0);
DBCharType* d_KEY_0lineitem__l_returnflag;
hipMalloc(&d_KEY_0lineitem__l_returnflag, sizeof(DBCharType) * COUNT0);
hipMemset(d_KEY_0lineitem__l_returnflag, 0, sizeof(DBCharType) * COUNT0);
DBCharType* d_KEY_0lineitem__l_linestatus;
hipMalloc(&d_KEY_0lineitem__l_linestatus, sizeof(DBCharType) * COUNT0);
hipMemset(d_KEY_0lineitem__l_linestatus, 0, sizeof(DBCharType) * COUNT0);
main_1<<<std::ceil((float)lineitem_size/128.), 128>>>(d_HT_0.ref(cuco::find), d_KEY_0lineitem__l_linestatus, d_KEY_0lineitem__l_returnflag, d_aggr0__tmp_attr0, d_aggr0__tmp_attr1, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_aggr0__tmp_attr9, d_aggr_rw__rw0, d_aggr_rw__rw1, d_aggr_rw__rw2, d_aggr_rw__rw3, d_aggr_rw__rw4, d_aggr_rw__rw5, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_linestatus, d_lineitem__l_quantity, d_lineitem__l_returnflag, d_lineitem__l_shipdate, d_lineitem__l_tax, lineitem_size);
//Materialize count
uint64_t* d_COUNT2;
hipMalloc(&d_COUNT2, sizeof(uint64_t));
hipMemset(d_COUNT2, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)COUNT0/128.), 128>>>(COUNT0, d_COUNT2);
uint64_t COUNT2;
hipMemcpy(&COUNT2, d_COUNT2, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX2;
hipMalloc(&d_MAT_IDX2, sizeof(uint64_t));
hipMemset(d_MAT_IDX2, 0, sizeof(uint64_t));
auto MAT2lineitem__l_returnflag = (DBCharType*)malloc(sizeof(DBCharType) * COUNT2);
DBCharType* d_MAT2lineitem__l_returnflag;
hipMalloc(&d_MAT2lineitem__l_returnflag, sizeof(DBCharType) * COUNT2);
auto MAT2lineitem__l_linestatus = (DBCharType*)malloc(sizeof(DBCharType) * COUNT2);
DBCharType* d_MAT2lineitem__l_linestatus;
hipMalloc(&d_MAT2lineitem__l_linestatus, sizeof(DBCharType) * COUNT2);
auto MAT2aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT2);
DBDecimalType* d_MAT2aggr0__tmp_attr0;
hipMalloc(&d_MAT2aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT2);
auto MAT2aggr0__tmp_attr1 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT2);
DBDecimalType* d_MAT2aggr0__tmp_attr1;
hipMalloc(&d_MAT2aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT2);
auto MAT2aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT2);
DBDecimalType* d_MAT2aggr0__tmp_attr2;
hipMalloc(&d_MAT2aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT2);
auto MAT2aggr0__tmp_attr4 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT2);
DBDecimalType* d_MAT2aggr0__tmp_attr4;
hipMalloc(&d_MAT2aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT2);
auto MAT2aggr0__tmp_attr6 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT2);
DBDecimalType* d_MAT2aggr0__tmp_attr6;
hipMalloc(&d_MAT2aggr0__tmp_attr6, sizeof(DBDecimalType) * COUNT2);
auto MAT2aggr0__tmp_attr7 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT2);
DBDecimalType* d_MAT2aggr0__tmp_attr7;
hipMalloc(&d_MAT2aggr0__tmp_attr7, sizeof(DBDecimalType) * COUNT2);
auto MAT2aggr0__tmp_attr8 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT2);
DBDecimalType* d_MAT2aggr0__tmp_attr8;
hipMalloc(&d_MAT2aggr0__tmp_attr8, sizeof(DBDecimalType) * COUNT2);
auto MAT2aggr0__tmp_attr9 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT2);
DBI64Type* d_MAT2aggr0__tmp_attr9;
hipMalloc(&d_MAT2aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT2);
main_3<<<std::ceil((float)COUNT0/128.), 128>>>(COUNT0, d_MAT2aggr0__tmp_attr0, d_MAT2aggr0__tmp_attr1, d_MAT2aggr0__tmp_attr2, d_MAT2aggr0__tmp_attr4, d_MAT2aggr0__tmp_attr6, d_MAT2aggr0__tmp_attr7, d_MAT2aggr0__tmp_attr8, d_MAT2aggr0__tmp_attr9, d_MAT2lineitem__l_linestatus, d_MAT2lineitem__l_returnflag, d_MAT_IDX2, d_aggr0__tmp_attr0, d_aggr0__tmp_attr1, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_aggr0__tmp_attr9, d_aggr_rw__rw0, d_aggr_rw__rw1, d_aggr_rw__rw2, d_aggr_rw__rw3, d_aggr_rw__rw4, d_aggr_rw__rw5, d_KEY_0lineitem__l_linestatus, d_KEY_0lineitem__l_returnflag);
hipMemcpy(MAT2lineitem__l_returnflag, d_MAT2lineitem__l_returnflag, sizeof(DBCharType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2lineitem__l_linestatus, d_MAT2lineitem__l_linestatus, sizeof(DBCharType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2aggr0__tmp_attr0, d_MAT2aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2aggr0__tmp_attr1, d_MAT2aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2aggr0__tmp_attr2, d_MAT2aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2aggr0__tmp_attr4, d_MAT2aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2aggr0__tmp_attr6, d_MAT2aggr0__tmp_attr6, sizeof(DBDecimalType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2aggr0__tmp_attr7, d_MAT2aggr0__tmp_attr7, sizeof(DBDecimalType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2aggr0__tmp_attr8, d_MAT2aggr0__tmp_attr8, sizeof(DBDecimalType) * COUNT2, hipMemcpyDeviceToHost);
hipMemcpy(MAT2aggr0__tmp_attr9, d_MAT2aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT2, hipMemcpyDeviceToHost);
auto end = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < COUNT2; i++) { std::cout << "" << MAT2lineitem__l_returnflag[i];
std::cout << "|" << MAT2lineitem__l_linestatus[i];
std::cout << "|" << MAT2aggr0__tmp_attr0[i];
std::cout << "|" << MAT2aggr0__tmp_attr1[i];
std::cout << "|" << MAT2aggr0__tmp_attr2[i];
std::cout << "|" << MAT2aggr0__tmp_attr4[i];
std::cout << "|" << MAT2aggr0__tmp_attr6[i];
std::cout << "|" << MAT2aggr0__tmp_attr7[i];
std::cout << "|" << MAT2aggr0__tmp_attr8[i];
std::cout << "|" << MAT2aggr0__tmp_attr9[i];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_KEY_0lineitem__l_linestatus);
hipFree(d_KEY_0lineitem__l_returnflag);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr0__tmp_attr1);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_aggr0__tmp_attr4);
hipFree(d_aggr0__tmp_attr9);
hipFree(d_aggr_rw__rw0);
hipFree(d_aggr_rw__rw1);
hipFree(d_aggr_rw__rw2);
hipFree(d_aggr_rw__rw3);
hipFree(d_aggr_rw__rw4);
hipFree(d_aggr_rw__rw5);
hipFree(d_COUNT2);
hipFree(d_MAT2aggr0__tmp_attr0);
hipFree(d_MAT2aggr0__tmp_attr1);
hipFree(d_MAT2aggr0__tmp_attr2);
hipFree(d_MAT2aggr0__tmp_attr4);
hipFree(d_MAT2aggr0__tmp_attr6);
hipFree(d_MAT2aggr0__tmp_attr7);
hipFree(d_MAT2aggr0__tmp_attr8);
hipFree(d_MAT2aggr0__tmp_attr9);
hipFree(d_MAT2lineitem__l_linestatus);
hipFree(d_MAT2lineitem__l_returnflag);
hipFree(d_MAT_IDX2);
free(MAT2aggr0__tmp_attr0);
free(MAT2aggr0__tmp_attr1);
free(MAT2aggr0__tmp_attr2);
free(MAT2aggr0__tmp_attr4);
free(MAT2aggr0__tmp_attr6);
free(MAT2aggr0__tmp_attr7);
free(MAT2aggr0__tmp_attr8);
free(MAT2aggr0__tmp_attr9);
free(MAT2lineitem__l_linestatus);
free(MAT2lineitem__l_returnflag);
}