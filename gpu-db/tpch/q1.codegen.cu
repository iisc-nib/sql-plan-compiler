#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
template<typename HASHTABLE_INSERT>
__global__ void count_5b52660ff6f0(HASHTABLE_INSERT HT_5b52660ca800, DBCharType* lineitem__l_linestatus, DBCharType* lineitem__l_returnflag, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 10471, Predicate::lte))) return;
uint64_t KEY_5b52660ca800 = 0;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];

KEY_5b52660ca800 |= reg_lineitem__l_returnflag;
auto reg_lineitem__l_linestatus = lineitem__l_linestatus[tid];
KEY_5b52660ca800 <<= 8;
KEY_5b52660ca800 |= reg_lineitem__l_linestatus;
//Create aggregation hash table
HT_5b52660ca800.insert(cuco::pair{KEY_5b52660ca800, 1});
}
template<typename HASHTABLE_FIND>
__global__ void main_5b52660ff6f0(HASHTABLE_FIND HT_5b52660ca800, DBCharType* KEY_5b52660ca800lineitem__l_linestatus, DBCharType* KEY_5b52660ca800lineitem__l_returnflag, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr0__tmp_attr1, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, DBI64Type* aggr0__tmp_attr9, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, DBDecimalType* aggr_rw__rw2, DBI64Type* aggr_rw__rw3, DBDecimalType* aggr_rw__rw4, DBI64Type* aggr_rw__rw5, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBCharType* lineitem__l_linestatus, DBDecimalType* lineitem__l_quantity, DBCharType* lineitem__l_returnflag, DBDateType* lineitem__l_shipdate, DBDecimalType* lineitem__l_tax, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 10471, Predicate::lte))) return;
uint64_t KEY_5b52660ca800 = 0;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];

KEY_5b52660ca800 |= reg_lineitem__l_returnflag;
auto reg_lineitem__l_linestatus = lineitem__l_linestatus[tid];
KEY_5b52660ca800 <<= 8;
KEY_5b52660ca800 |= reg_lineitem__l_linestatus;
//Aggregate in hashtable
auto buf_idx_5b52660ca800 = HT_5b52660ca800.find(KEY_5b52660ca800)->second;
aggregate_sum(&aggr0__tmp_attr9[buf_idx_5b52660ca800], 1);
auto reg_lineitem__l_tax = lineitem__l_tax[tid];
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr5 = ((reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount))) * ((1) + (reg_lineitem__l_tax));
aggregate_sum(&aggr0__tmp_attr4[buf_idx_5b52660ca800], reg_map0__tmp_attr5);
auto reg_map0__tmp_attr3 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_5b52660ca800], reg_map0__tmp_attr3);
aggregate_sum(&aggr0__tmp_attr1[buf_idx_5b52660ca800], reg_lineitem__l_extendedprice);
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5b52660ca800], reg_lineitem__l_quantity);
aggregate_sum(&aggr_rw__rw0[buf_idx_5b52660ca800], reg_lineitem__l_discount);
aggregate_sum(&aggr_rw__rw1[buf_idx_5b52660ca800], 1);
aggregate_sum(&aggr_rw__rw2[buf_idx_5b52660ca800], reg_lineitem__l_extendedprice);
aggregate_sum(&aggr_rw__rw3[buf_idx_5b52660ca800], 1);
aggregate_sum(&aggr_rw__rw4[buf_idx_5b52660ca800], reg_lineitem__l_quantity);
aggregate_sum(&aggr_rw__rw5[buf_idx_5b52660ca800], 1);
KEY_5b52660ca800lineitem__l_returnflag[buf_idx_5b52660ca800] = reg_lineitem__l_returnflag;
KEY_5b52660ca800lineitem__l_linestatus[buf_idx_5b52660ca800] = reg_lineitem__l_linestatus;
}
__global__ void count_5b52660ff930(uint64_t* COUNT5b52660ac350, size_t COUNT5b52660ca800) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5b52660ca800) return;
//Materialize count
atomicAdd((int*)COUNT5b52660ac350, 1);
}
__global__ void main_5b52660ff930(size_t COUNT5b52660ca800, DBDecimalType* MAT5b52660ac350aggr0__tmp_attr0, DBDecimalType* MAT5b52660ac350aggr0__tmp_attr1, DBDecimalType* MAT5b52660ac350aggr0__tmp_attr2, DBDecimalType* MAT5b52660ac350aggr0__tmp_attr4, DBDecimalType* MAT5b52660ac350aggr0__tmp_attr6, DBDecimalType* MAT5b52660ac350aggr0__tmp_attr7, DBDecimalType* MAT5b52660ac350aggr0__tmp_attr8, DBI64Type* MAT5b52660ac350aggr0__tmp_attr9, DBCharType* MAT5b52660ac350lineitem__l_linestatus, DBCharType* MAT5b52660ac350lineitem__l_returnflag, uint64_t* MAT_IDX5b52660ac350, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr0__tmp_attr1, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, DBI64Type* aggr0__tmp_attr9, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, DBDecimalType* aggr_rw__rw2, DBI64Type* aggr_rw__rw3, DBDecimalType* aggr_rw__rw4, DBI64Type* aggr_rw__rw5, DBCharType* lineitem__l_linestatus, DBCharType* lineitem__l_returnflag) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5b52660ca800) return;
//Materialize buffers
auto mat_idx5b52660ac350 = atomicAdd((int*)MAT_IDX5b52660ac350, 1);
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];
MAT5b52660ac350lineitem__l_returnflag[mat_idx5b52660ac350] = reg_lineitem__l_returnflag;
auto reg_lineitem__l_linestatus = lineitem__l_linestatus[tid];
MAT5b52660ac350lineitem__l_linestatus[mat_idx5b52660ac350] = reg_lineitem__l_linestatus;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5b52660ac350aggr0__tmp_attr0[mat_idx5b52660ac350] = reg_aggr0__tmp_attr0;
auto reg_aggr0__tmp_attr1 = aggr0__tmp_attr1[tid];
MAT5b52660ac350aggr0__tmp_attr1[mat_idx5b52660ac350] = reg_aggr0__tmp_attr1;
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT5b52660ac350aggr0__tmp_attr2[mat_idx5b52660ac350] = reg_aggr0__tmp_attr2;
auto reg_aggr0__tmp_attr4 = aggr0__tmp_attr4[tid];
MAT5b52660ac350aggr0__tmp_attr4[mat_idx5b52660ac350] = reg_aggr0__tmp_attr4;
auto reg_aggr_rw__rw5 = aggr_rw__rw5[tid];
auto reg_aggr_rw__rw4 = aggr_rw__rw4[tid];
auto reg_aggr0__tmp_attr6 = (reg_aggr_rw__rw4) / ((DBDecimalType)(reg_aggr_rw__rw5));
MAT5b52660ac350aggr0__tmp_attr6[mat_idx5b52660ac350] = reg_aggr0__tmp_attr6;
auto reg_aggr_rw__rw3 = aggr_rw__rw3[tid];
auto reg_aggr_rw__rw2 = aggr_rw__rw2[tid];
auto reg_aggr0__tmp_attr7 = (reg_aggr_rw__rw2) / ((DBDecimalType)(reg_aggr_rw__rw3));
MAT5b52660ac350aggr0__tmp_attr7[mat_idx5b52660ac350] = reg_aggr0__tmp_attr7;
auto reg_aggr_rw__rw1 = aggr_rw__rw1[tid];
auto reg_aggr_rw__rw0 = aggr_rw__rw0[tid];
auto reg_aggr0__tmp_attr8 = (reg_aggr_rw__rw0) / ((DBDecimalType)(reg_aggr_rw__rw1));
MAT5b52660ac350aggr0__tmp_attr8[mat_idx5b52660ac350] = reg_aggr0__tmp_attr8;
auto reg_aggr0__tmp_attr9 = aggr0__tmp_attr9[tid];
MAT5b52660ac350aggr0__tmp_attr9[mat_idx5b52660ac350] = reg_aggr0__tmp_attr9;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map) {
//Create aggregation hash table
auto d_HT_5b52660ca800 = cuco::static_map{ (int)5930889*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5b52660ff6f0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_HT_5b52660ca800.ref(cuco::insert), d_lineitem__l_linestatus, d_lineitem__l_returnflag, d_lineitem__l_shipdate, lineitem_size);
size_t COUNT5b52660ca800 = d_HT_5b52660ca800.size();
thrust::device_vector<int64_t> keys_5b52660ca800(COUNT5b52660ca800), vals_5b52660ca800(COUNT5b52660ca800);
d_HT_5b52660ca800.retrieve_all(keys_5b52660ca800.begin(), vals_5b52660ca800.begin());
d_HT_5b52660ca800.clear();
int64_t* raw_keys5b52660ca800 = thrust::raw_pointer_cast(keys_5b52660ca800.data());
insertKeys<<<std::ceil((float)COUNT5b52660ca800/32.), 32>>>(raw_keys5b52660ca800, d_HT_5b52660ca800.ref(cuco::insert), COUNT5b52660ca800);
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr9;
hipMalloc(&d_aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT5b52660ca800);
hipMemset(d_aggr0__tmp_attr9, 0, sizeof(DBI64Type) * COUNT5b52660ca800);
DBDecimalType* d_aggr0__tmp_attr4;
hipMalloc(&d_aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT5b52660ca800);
hipMemset(d_aggr0__tmp_attr4, 0, sizeof(DBDecimalType) * COUNT5b52660ca800);
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT5b52660ca800);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT5b52660ca800);
DBDecimalType* d_aggr0__tmp_attr1;
hipMalloc(&d_aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT5b52660ca800);
hipMemset(d_aggr0__tmp_attr1, 0, sizeof(DBDecimalType) * COUNT5b52660ca800);
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5b52660ca800);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5b52660ca800);
DBDecimalType* d_aggr_rw__rw0;
hipMalloc(&d_aggr_rw__rw0, sizeof(DBDecimalType) * COUNT5b52660ca800);
hipMemset(d_aggr_rw__rw0, 0, sizeof(DBDecimalType) * COUNT5b52660ca800);
DBI64Type* d_aggr_rw__rw1;
hipMalloc(&d_aggr_rw__rw1, sizeof(DBI64Type) * COUNT5b52660ca800);
hipMemset(d_aggr_rw__rw1, 0, sizeof(DBI64Type) * COUNT5b52660ca800);
DBDecimalType* d_aggr_rw__rw2;
hipMalloc(&d_aggr_rw__rw2, sizeof(DBDecimalType) * COUNT5b52660ca800);
hipMemset(d_aggr_rw__rw2, 0, sizeof(DBDecimalType) * COUNT5b52660ca800);
DBI64Type* d_aggr_rw__rw3;
hipMalloc(&d_aggr_rw__rw3, sizeof(DBI64Type) * COUNT5b52660ca800);
hipMemset(d_aggr_rw__rw3, 0, sizeof(DBI64Type) * COUNT5b52660ca800);
DBDecimalType* d_aggr_rw__rw4;
hipMalloc(&d_aggr_rw__rw4, sizeof(DBDecimalType) * COUNT5b52660ca800);
hipMemset(d_aggr_rw__rw4, 0, sizeof(DBDecimalType) * COUNT5b52660ca800);
DBI64Type* d_aggr_rw__rw5;
hipMalloc(&d_aggr_rw__rw5, sizeof(DBI64Type) * COUNT5b52660ca800);
hipMemset(d_aggr_rw__rw5, 0, sizeof(DBI64Type) * COUNT5b52660ca800);
DBCharType* d_KEY_5b52660ca800lineitem__l_returnflag;
hipMalloc(&d_KEY_5b52660ca800lineitem__l_returnflag, sizeof(DBCharType) * COUNT5b52660ca800);
hipMemset(d_KEY_5b52660ca800lineitem__l_returnflag, 0, sizeof(DBCharType) * COUNT5b52660ca800);
DBCharType* d_KEY_5b52660ca800lineitem__l_linestatus;
hipMalloc(&d_KEY_5b52660ca800lineitem__l_linestatus, sizeof(DBCharType) * COUNT5b52660ca800);
hipMemset(d_KEY_5b52660ca800lineitem__l_linestatus, 0, sizeof(DBCharType) * COUNT5b52660ca800);
main_5b52660ff6f0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_HT_5b52660ca800.ref(cuco::find), d_KEY_5b52660ca800lineitem__l_linestatus, d_KEY_5b52660ca800lineitem__l_returnflag, d_aggr0__tmp_attr0, d_aggr0__tmp_attr1, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_aggr0__tmp_attr9, d_aggr_rw__rw0, d_aggr_rw__rw1, d_aggr_rw__rw2, d_aggr_rw__rw3, d_aggr_rw__rw4, d_aggr_rw__rw5, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_linestatus, d_lineitem__l_quantity, d_lineitem__l_returnflag, d_lineitem__l_shipdate, d_lineitem__l_tax, lineitem_size);
//Materialize count
uint64_t* d_COUNT5b52660ac350;
hipMalloc(&d_COUNT5b52660ac350, sizeof(uint64_t));
hipMemset(d_COUNT5b52660ac350, 0, sizeof(uint64_t));
count_5b52660ff930<<<std::ceil((float)COUNT5b52660ca800/32.), 32>>>(d_COUNT5b52660ac350, COUNT5b52660ca800);
uint64_t COUNT5b52660ac350;
hipMemcpy(&COUNT5b52660ac350, d_COUNT5b52660ac350, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5b52660ac350;
hipMalloc(&d_MAT_IDX5b52660ac350, sizeof(uint64_t));
hipMemset(d_MAT_IDX5b52660ac350, 0, sizeof(uint64_t));
auto MAT5b52660ac350lineitem__l_returnflag = (DBCharType*)malloc(sizeof(DBCharType) * COUNT5b52660ac350);
DBCharType* d_MAT5b52660ac350lineitem__l_returnflag;
hipMalloc(&d_MAT5b52660ac350lineitem__l_returnflag, sizeof(DBCharType) * COUNT5b52660ac350);
auto MAT5b52660ac350lineitem__l_linestatus = (DBCharType*)malloc(sizeof(DBCharType) * COUNT5b52660ac350);
DBCharType* d_MAT5b52660ac350lineitem__l_linestatus;
hipMalloc(&d_MAT5b52660ac350lineitem__l_linestatus, sizeof(DBCharType) * COUNT5b52660ac350);
auto MAT5b52660ac350aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5b52660ac350);
DBDecimalType* d_MAT5b52660ac350aggr0__tmp_attr0;
hipMalloc(&d_MAT5b52660ac350aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5b52660ac350);
auto MAT5b52660ac350aggr0__tmp_attr1 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5b52660ac350);
DBDecimalType* d_MAT5b52660ac350aggr0__tmp_attr1;
hipMalloc(&d_MAT5b52660ac350aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT5b52660ac350);
auto MAT5b52660ac350aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5b52660ac350);
DBDecimalType* d_MAT5b52660ac350aggr0__tmp_attr2;
hipMalloc(&d_MAT5b52660ac350aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT5b52660ac350);
auto MAT5b52660ac350aggr0__tmp_attr4 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5b52660ac350);
DBDecimalType* d_MAT5b52660ac350aggr0__tmp_attr4;
hipMalloc(&d_MAT5b52660ac350aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT5b52660ac350);
auto MAT5b52660ac350aggr0__tmp_attr6 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5b52660ac350);
DBDecimalType* d_MAT5b52660ac350aggr0__tmp_attr6;
hipMalloc(&d_MAT5b52660ac350aggr0__tmp_attr6, sizeof(DBDecimalType) * COUNT5b52660ac350);
auto MAT5b52660ac350aggr0__tmp_attr7 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5b52660ac350);
DBDecimalType* d_MAT5b52660ac350aggr0__tmp_attr7;
hipMalloc(&d_MAT5b52660ac350aggr0__tmp_attr7, sizeof(DBDecimalType) * COUNT5b52660ac350);
auto MAT5b52660ac350aggr0__tmp_attr8 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5b52660ac350);
DBDecimalType* d_MAT5b52660ac350aggr0__tmp_attr8;
hipMalloc(&d_MAT5b52660ac350aggr0__tmp_attr8, sizeof(DBDecimalType) * COUNT5b52660ac350);
auto MAT5b52660ac350aggr0__tmp_attr9 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT5b52660ac350);
DBI64Type* d_MAT5b52660ac350aggr0__tmp_attr9;
hipMalloc(&d_MAT5b52660ac350aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT5b52660ac350);
main_5b52660ff930<<<std::ceil((float)COUNT5b52660ca800/32.), 32>>>(COUNT5b52660ca800, d_MAT5b52660ac350aggr0__tmp_attr0, d_MAT5b52660ac350aggr0__tmp_attr1, d_MAT5b52660ac350aggr0__tmp_attr2, d_MAT5b52660ac350aggr0__tmp_attr4, d_MAT5b52660ac350aggr0__tmp_attr6, d_MAT5b52660ac350aggr0__tmp_attr7, d_MAT5b52660ac350aggr0__tmp_attr8, d_MAT5b52660ac350aggr0__tmp_attr9, d_MAT5b52660ac350lineitem__l_linestatus, d_MAT5b52660ac350lineitem__l_returnflag, d_MAT_IDX5b52660ac350, d_aggr0__tmp_attr0, d_aggr0__tmp_attr1, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_aggr0__tmp_attr9, d_aggr_rw__rw0, d_aggr_rw__rw1, d_aggr_rw__rw2, d_aggr_rw__rw3, d_aggr_rw__rw4, d_aggr_rw__rw5, d_KEY_5b52660ca800lineitem__l_linestatus, d_KEY_5b52660ca800lineitem__l_returnflag);
hipMemcpy(MAT5b52660ac350lineitem__l_returnflag, d_MAT5b52660ac350lineitem__l_returnflag, sizeof(DBCharType) * COUNT5b52660ac350, hipMemcpyDeviceToHost);
hipMemcpy(MAT5b52660ac350lineitem__l_linestatus, d_MAT5b52660ac350lineitem__l_linestatus, sizeof(DBCharType) * COUNT5b52660ac350, hipMemcpyDeviceToHost);
hipMemcpy(MAT5b52660ac350aggr0__tmp_attr0, d_MAT5b52660ac350aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5b52660ac350, hipMemcpyDeviceToHost);
hipMemcpy(MAT5b52660ac350aggr0__tmp_attr1, d_MAT5b52660ac350aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT5b52660ac350, hipMemcpyDeviceToHost);
hipMemcpy(MAT5b52660ac350aggr0__tmp_attr2, d_MAT5b52660ac350aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT5b52660ac350, hipMemcpyDeviceToHost);
hipMemcpy(MAT5b52660ac350aggr0__tmp_attr4, d_MAT5b52660ac350aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT5b52660ac350, hipMemcpyDeviceToHost);
hipMemcpy(MAT5b52660ac350aggr0__tmp_attr6, d_MAT5b52660ac350aggr0__tmp_attr6, sizeof(DBDecimalType) * COUNT5b52660ac350, hipMemcpyDeviceToHost);
hipMemcpy(MAT5b52660ac350aggr0__tmp_attr7, d_MAT5b52660ac350aggr0__tmp_attr7, sizeof(DBDecimalType) * COUNT5b52660ac350, hipMemcpyDeviceToHost);
hipMemcpy(MAT5b52660ac350aggr0__tmp_attr8, d_MAT5b52660ac350aggr0__tmp_attr8, sizeof(DBDecimalType) * COUNT5b52660ac350, hipMemcpyDeviceToHost);
hipMemcpy(MAT5b52660ac350aggr0__tmp_attr9, d_MAT5b52660ac350aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT5b52660ac350, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5b52660ac350; i++) { std::cout << MAT5b52660ac350lineitem__l_returnflag[i] << "\t";
std::cout << MAT5b52660ac350lineitem__l_linestatus[i] << "\t";
std::cout << MAT5b52660ac350aggr0__tmp_attr0[i] << "\t";
std::cout << MAT5b52660ac350aggr0__tmp_attr1[i] << "\t";
std::cout << MAT5b52660ac350aggr0__tmp_attr2[i] << "\t";
std::cout << MAT5b52660ac350aggr0__tmp_attr4[i] << "\t";
std::cout << MAT5b52660ac350aggr0__tmp_attr6[i] << "\t";
std::cout << MAT5b52660ac350aggr0__tmp_attr7[i] << "\t";
std::cout << MAT5b52660ac350aggr0__tmp_attr8[i] << "\t";
std::cout << MAT5b52660ac350aggr0__tmp_attr9[i] << "\t";
std::cout << std::endl; }
hipFree(d_KEY_5b52660ca800lineitem__l_linestatus);
hipFree(d_KEY_5b52660ca800lineitem__l_returnflag);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr0__tmp_attr1);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_aggr0__tmp_attr4);
hipFree(d_aggr0__tmp_attr9);
hipFree(d_aggr_rw__rw0);
hipFree(d_aggr_rw__rw1);
hipFree(d_aggr_rw__rw2);
hipFree(d_aggr_rw__rw3);
hipFree(d_aggr_rw__rw4);
hipFree(d_aggr_rw__rw5);
hipFree(d_COUNT5b52660ac350);
hipFree(d_MAT5b52660ac350aggr0__tmp_attr0);
hipFree(d_MAT5b52660ac350aggr0__tmp_attr1);
hipFree(d_MAT5b52660ac350aggr0__tmp_attr2);
hipFree(d_MAT5b52660ac350aggr0__tmp_attr4);
hipFree(d_MAT5b52660ac350aggr0__tmp_attr6);
hipFree(d_MAT5b52660ac350aggr0__tmp_attr7);
hipFree(d_MAT5b52660ac350aggr0__tmp_attr8);
hipFree(d_MAT5b52660ac350aggr0__tmp_attr9);
hipFree(d_MAT5b52660ac350lineitem__l_linestatus);
hipFree(d_MAT5b52660ac350lineitem__l_returnflag);
hipFree(d_MAT_IDX5b52660ac350);
free(MAT5b52660ac350aggr0__tmp_attr0);
free(MAT5b52660ac350aggr0__tmp_attr1);
free(MAT5b52660ac350aggr0__tmp_attr2);
free(MAT5b52660ac350aggr0__tmp_attr4);
free(MAT5b52660ac350aggr0__tmp_attr6);
free(MAT5b52660ac350aggr0__tmp_attr7);
free(MAT5b52660ac350aggr0__tmp_attr8);
free(MAT5b52660ac350aggr0__tmp_attr9);
free(MAT5b52660ac350lineitem__l_linestatus);
free(MAT5b52660ac350lineitem__l_returnflag);
}