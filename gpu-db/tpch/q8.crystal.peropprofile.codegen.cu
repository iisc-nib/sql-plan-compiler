#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_1(uint64_t* COUNT2, DBStringType* part__p_type, size_t part_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_part__p_type[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_type[ITEM] = part__p_type[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_part__p_type[ITEM], "ECONOMY ANODIZED STEEL", Predicate::eq);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT2, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_INSERT HT_2, int64_t* cycles_per_warp_main_1_join_build_2, int64_t* cycles_per_warp_main_1_selection_0, DBI32Type* part__p_partkey, DBStringType* part__p_type, size_t part_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_part__p_type[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_type[ITEM] = part__p_type[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_part__p_type[ITEM], "ECONOMY ANODIZED STEEL", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_2 = atomicAdd((int*)BUF_IDX_2, 1);
HT_2.insert(cuco::pair{KEY_2[ITEM], buf_idx_2});
BUF_2[(buf_idx_2) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_build_2[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_5(uint64_t* COUNT6, DBStringType* region__r_name, size_t region_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_region__r_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
reg_region__r_name[ITEM] = region__r_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_region__r_name[ITEM], "AMERICA", Predicate::eq);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT6, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_INSERT HT_6, int64_t* cycles_per_warp_main_5_join_build_6, int64_t* cycles_per_warp_main_5_selection_4, DBStringType* region__r_name, DBI32Type* region__r_regionkey, size_t region_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_region__r_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
reg_region__r_name[ITEM] = region__r_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_region__r_name[ITEM], "AMERICA", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_selection_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_region__r_regionkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
reg_region__r_regionkey[ITEM] = region__r_regionkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_region__r_regionkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_6 = atomicAdd((int*)BUF_IDX_6, 1);
HT_6.insert(cuco::pair{KEY_6[ITEM], buf_idx_6});
BUF_6[(buf_idx_6) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_join_build_6[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_9(uint64_t* COUNT10, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_orders__o_orderdate[ITEM], 9131, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate[ITEM], 9861, Predicate::lte);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT10, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_9(uint64_t* BUF_10, uint64_t* BUF_IDX_10, HASHTABLE_INSERT HT_10, int64_t* cycles_per_warp_main_9_join_build_10, int64_t* cycles_per_warp_main_9_selection_8, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_orders__o_orderdate[ITEM], 9131, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate[ITEM], 9861, Predicate::lte);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_9_selection_8[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_orders__o_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_10 = atomicAdd((int*)BUF_IDX_10, 1);
HT_10.insert(cuco::pair{KEY_10[ITEM], buf_idx_10});
BUF_10[(buf_idx_10) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_9_join_build_10[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_12(uint64_t* COUNT11, size_t customer_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT11, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_12(uint64_t* BUF_11, uint64_t* BUF_IDX_11, HASHTABLE_INSERT HT_11, DBI32Type* customer__c_custkey, size_t customer_size, int64_t* cycles_per_warp_main_12_join_build_11) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_11[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_11[ITEM] = 0;
KEY_11[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_11 = atomicAdd((int*)BUF_IDX_11, 1);
HT_11.insert(cuco::pair{KEY_11[ITEM], buf_idx_11});
BUF_11[(buf_idx_11) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_12_join_build_11[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE>
__global__ void count_7(uint64_t* BUF_6, uint64_t* COUNT13, HASHTABLE_PROBE HT_6, DBI32Type* n1___n_regionkey, size_t nation_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_n1___n_regionkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n1___n_regionkey[ITEM] = n1___n_regionkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_n1___n_regionkey[ITEM];
}
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT13, 1);
}
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_7(uint64_t* BUF_13, uint64_t* BUF_6, uint64_t* BUF_IDX_13, HASHTABLE_INSERT HT_13, HASHTABLE_PROBE HT_6, int64_t* cycles_per_warp_main_7_join_build_13, int64_t* cycles_per_warp_main_7_join_probe_6, DBI32Type* n1___n_nationkey, DBI32Type* n1___n_regionkey, size_t nation_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_n1___n_regionkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n1___n_regionkey[ITEM] = n1___n_regionkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_n1___n_regionkey[ITEM];
}
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_probe_6[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_13[ITEMS_PER_THREAD];
DBI32Type reg_n1___n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n1___n_nationkey[ITEM] = n1___n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_13[ITEM] = 0;
KEY_13[ITEM] |= reg_n1___n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_13 = atomicAdd((int*)BUF_IDX_13, 1);
HT_13.insert(cuco::pair{KEY_13[ITEM], buf_idx_13});
BUF_13[(buf_idx_13) * 2 + 0] = BUF_6[slot_second6[ITEM] * 1 + 0];
BUF_13[(buf_idx_13) * 2 + 1] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_build_13[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_15(uint64_t* COUNT14, size_t supplier_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT14, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_15(uint64_t* BUF_14, uint64_t* BUF_IDX_14, HASHTABLE_INSERT HT_14, int64_t* cycles_per_warp_main_15_join_build_14, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_14[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_14[ITEM] = 0;
KEY_14[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_14 = atomicAdd((int*)BUF_IDX_14, 1);
HT_14.insert(cuco::pair{KEY_14[ITEM], buf_idx_14});
BUF_14[(buf_idx_14) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_15_join_build_14[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_17(uint64_t* COUNT16, size_t nation_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT16, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_17(uint64_t* BUF_16, uint64_t* BUF_IDX_16, HASHTABLE_INSERT HT_16, int64_t* cycles_per_warp_main_17_join_build_16, DBI32Type* n2___n_nationkey, size_t nation_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_16[ITEMS_PER_THREAD];
DBI32Type reg_n2___n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n2___n_nationkey[ITEM] = n2___n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_16[ITEM] = 0;
KEY_16[ITEM] |= reg_n2___n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_16 = atomicAdd((int*)BUF_IDX_16, 1);
HT_16.insert(cuco::pair{KEY_16[ITEM], buf_idx_16});
BUF_16[(buf_idx_16) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_17_join_build_16[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_3(uint64_t* BUF_10, uint64_t* BUF_11, uint64_t* BUF_13, uint64_t* BUF_14, uint64_t* BUF_16, uint64_t* BUF_2, HASHTABLE_PROBE HT_10, HASHTABLE_PROBE HT_11, HASHTABLE_PROBE HT_13, HASHTABLE_PROBE HT_14, HASHTABLE_PROBE HT_16, HASHTABLE_PROBE HT_2, HASHTABLE_INSERT HT_21, DBI32Type* customer__c_nationkey, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* supplier__s_nationkey) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
int64_t slot_second2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_2 = HT_2.find(KEY_2[ITEM]);
if (SLOT_2 == HT_2.end()) {selection_flags[ITEM] = 0; continue;}
slot_second2[ITEM] = SLOT_2->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
int64_t slot_second10[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_10 = HT_10.find(KEY_10[ITEM]);
if (SLOT_10 == HT_10.end()) {selection_flags[ITEM] = 0; continue;}
slot_second10[ITEM] = SLOT_10->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_11[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_custkey[ITEM] = orders__o_custkey[BUF_10[slot_second10[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_11[ITEM] = 0;
KEY_11[ITEM] |= reg_orders__o_custkey[ITEM];
}
int64_t slot_second11[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_11 = HT_11.find(KEY_11[ITEM]);
if (SLOT_11 == HT_11.end()) {selection_flags[ITEM] = 0; continue;}
slot_second11[ITEM] = SLOT_11->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_13[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[BUF_11[slot_second11[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_13[ITEM] = 0;
KEY_13[ITEM] |= reg_customer__c_nationkey[ITEM];
}
int64_t slot_second13[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_13 = HT_13.find(KEY_13[ITEM]);
if (SLOT_13 == HT_13.end()) {selection_flags[ITEM] = 0; continue;}
slot_second13[ITEM] = SLOT_13->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_14[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_suppkey[ITEM] = lineitem__l_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_14[ITEM] = 0;
KEY_14[ITEM] |= reg_lineitem__l_suppkey[ITEM];
}
int64_t slot_second14[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_14 = HT_14.find(KEY_14[ITEM]);
if (SLOT_14 == HT_14.end()) {selection_flags[ITEM] = 0; continue;}
slot_second14[ITEM] = SLOT_14->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_16[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_supplier__s_nationkey[ITEM] = supplier__s_nationkey[BUF_14[slot_second14[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_16[ITEM] = 0;
KEY_16[ITEM] |= reg_supplier__s_nationkey[ITEM];
}
int64_t slot_second16[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_16 = HT_16.find(KEY_16[ITEM]);
if (SLOT_16 == HT_16.end()) {selection_flags[ITEM] = 0; continue;}
slot_second16[ITEM] = SLOT_16->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[BUF_10[slot_second10[ITEM] * 1 + 0]];
}
uint64_t KEY_21[ITEMS_PER_THREAD];
DBI64Type reg_map0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr0[ITEM] = ExtractFromDate("year", reg_orders__o_orderdate[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_21[ITEM] = 0;
KEY_21[ITEM] |= (DBI32Type)reg_map0__tmp_attr0[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_21.insert(cuco::pair{KEY_21[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_3(uint64_t* BUF_10, uint64_t* BUF_11, uint64_t* BUF_13, uint64_t* BUF_14, uint64_t* BUF_16, uint64_t* BUF_2, HASHTABLE_PROBE HT_10, HASHTABLE_PROBE HT_11, HASHTABLE_PROBE HT_13, HASHTABLE_PROBE HT_14, HASHTABLE_PROBE HT_16, HASHTABLE_PROBE HT_2, HASHTABLE_FIND HT_21, DBI64Type* KEY_21map0__tmp_attr0, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, DBI32Type* customer__c_nationkey, int64_t* cycles_per_warp_main_3_aggregation_21, int64_t* cycles_per_warp_main_3_join_probe_10, int64_t* cycles_per_warp_main_3_join_probe_11, int64_t* cycles_per_warp_main_3_join_probe_13, int64_t* cycles_per_warp_main_3_join_probe_14, int64_t* cycles_per_warp_main_3_join_probe_16, int64_t* cycles_per_warp_main_3_join_probe_2, int64_t* cycles_per_warp_main_3_map_18, int64_t* cycles_per_warp_main_3_map_19, int64_t* cycles_per_warp_main_3_map_20, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n2___n_name, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* supplier__s_nationkey) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
int64_t slot_second2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_2 = HT_2.find(KEY_2[ITEM]);
if (SLOT_2 == HT_2.end()) {selection_flags[ITEM] = 0; continue;}
slot_second2[ITEM] = SLOT_2->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_probe_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
int64_t slot_second10[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_10 = HT_10.find(KEY_10[ITEM]);
if (SLOT_10 == HT_10.end()) {selection_flags[ITEM] = 0; continue;}
slot_second10[ITEM] = SLOT_10->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_probe_10[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_11[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_custkey[ITEM] = orders__o_custkey[BUF_10[slot_second10[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_11[ITEM] = 0;
KEY_11[ITEM] |= reg_orders__o_custkey[ITEM];
}
int64_t slot_second11[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_11 = HT_11.find(KEY_11[ITEM]);
if (SLOT_11 == HT_11.end()) {selection_flags[ITEM] = 0; continue;}
slot_second11[ITEM] = SLOT_11->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_probe_11[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_13[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[BUF_11[slot_second11[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_13[ITEM] = 0;
KEY_13[ITEM] |= reg_customer__c_nationkey[ITEM];
}
int64_t slot_second13[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_13 = HT_13.find(KEY_13[ITEM]);
if (SLOT_13 == HT_13.end()) {selection_flags[ITEM] = 0; continue;}
slot_second13[ITEM] = SLOT_13->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_probe_13[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_14[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_suppkey[ITEM] = lineitem__l_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_14[ITEM] = 0;
KEY_14[ITEM] |= reg_lineitem__l_suppkey[ITEM];
}
int64_t slot_second14[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_14 = HT_14.find(KEY_14[ITEM]);
if (SLOT_14 == HT_14.end()) {selection_flags[ITEM] = 0; continue;}
slot_second14[ITEM] = SLOT_14->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_probe_14[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_16[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_supplier__s_nationkey[ITEM] = supplier__s_nationkey[BUF_14[slot_second14[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_16[ITEM] = 0;
KEY_16[ITEM] |= reg_supplier__s_nationkey[ITEM];
}
int64_t slot_second16[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_16 = HT_16.find(KEY_16[ITEM]);
if (SLOT_16 == HT_16.end()) {selection_flags[ITEM] = 0; continue;}
slot_second16[ITEM] = SLOT_16->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_probe_16[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_map_18[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[BUF_10[slot_second10[ITEM] * 1 + 0]];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_map_19[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_map_20[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_21[ITEMS_PER_THREAD];
DBI64Type reg_map0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr0[ITEM] = ExtractFromDate("year", reg_orders__o_orderdate[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_21[ITEM] = 0;
KEY_21[ITEM] |= (DBI32Type)reg_map0__tmp_attr0[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]));
}
DBStringType reg_n2___n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_n2___n_name[ITEM] = n2___n_name[BUF_16[slot_second16[ITEM] * 1 + 0]];
}
DBDecimalType reg_map1__tmp_attr3[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map1__tmp_attr3[ITEM] = ((evaluatePredicate(reg_n2___n_name[ITEM], "BRAZIL", Predicate::eq))) ? ((reg_map0__tmp_attr1[ITEM])) : ((0.0));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_21 = HT_21.find(KEY_21[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr4[buf_idx_21], reg_map0__tmp_attr1[ITEM]);
aggregate_sum(&aggr0__tmp_attr2[buf_idx_21], reg_map1__tmp_attr3[ITEM]);
KEY_21map0__tmp_attr0[buf_idx_21] = reg_map0__tmp_attr0[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_aggregation_21[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_23(size_t COUNT21, uint64_t* COUNT24) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT21); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT24, 1);
}
}
__global__ void main_23(size_t COUNT21, DBI64Type* MAT24map0__tmp_attr0, DBDecimalType* MAT24map2__tmp_attr5, uint64_t* MAT_IDX24, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, int64_t* cycles_per_warp_main_23_map_22, int64_t* cycles_per_warp_main_23_materialize_24, DBI64Type* map0__tmp_attr0) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_23_map_22[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBI64Type reg_map0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT21); ++ITEM) {
reg_map0__tmp_attr0[ITEM] = map0__tmp_attr0[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT21); ++ITEM) {
reg_aggr0__tmp_attr4[ITEM] = aggr0__tmp_attr4[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT21); ++ITEM) {
reg_aggr0__tmp_attr2[ITEM] = aggr0__tmp_attr2[ITEM*TB + tid];
}
DBDecimalType reg_map2__tmp_attr5[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT21); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map2__tmp_attr5[ITEM] = (reg_aggr0__tmp_attr2[ITEM]) / (reg_aggr0__tmp_attr4[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT21); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx24 = atomicAdd((int*)MAT_IDX24, 1);
MAT24map0__tmp_attr0[mat_idx24] = reg_map0__tmp_attr0[ITEM];
MAT24map2__tmp_attr5[mat_idx24] = reg_map2__tmp_attr5[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_23_materialize_24[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
//Materialize count
uint64_t* d_COUNT2;
hipMalloc(&d_COUNT2, sizeof(uint64_t));
hipMemset(d_COUNT2, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)part_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT2, d_part__p_type, part_size);
uint64_t COUNT2;
hipMemcpy(&COUNT2, d_COUNT2, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_1_join_build_2;
auto main_1_join_build_2_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_build_2, sizeof(int64_t) * main_1_join_build_2_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_build_2, -1, sizeof(int64_t) * main_1_join_build_2_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 1);
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)part_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_2, d_BUF_IDX_2, d_HT_2.ref(cuco::insert), d_cycles_per_warp_main_1_join_build_2, d_cycles_per_warp_main_1_selection_0, d_part__p_partkey, d_part__p_type, part_size);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_build_2 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_build_2_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_build_2, d_cycles_per_warp_main_1_join_build_2, sizeof(int64_t) * main_1_join_build_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_build_2 ";
for (auto i=0ull; i < main_1_join_build_2_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_build_2[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_5_selection_4;
auto main_5_selection_4_cpw_size = std::ceil((float)region_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_selection_4, sizeof(int64_t) * main_5_selection_4_cpw_size);
hipMemset(d_cycles_per_warp_main_5_selection_4, -1, sizeof(int64_t) * main_5_selection_4_cpw_size);
//Materialize count
uint64_t* d_COUNT6;
hipMalloc(&d_COUNT6, sizeof(uint64_t));
hipMemset(d_COUNT6, 0, sizeof(uint64_t));
count_5<<<std::ceil((float)region_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT6, d_region__r_name, region_size);
uint64_t COUNT6;
hipMemcpy(&COUNT6, d_COUNT6, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_5_join_build_6;
auto main_5_join_build_6_cpw_size = std::ceil((float)region_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_join_build_6, sizeof(int64_t) * main_5_join_build_6_cpw_size);
hipMemset(d_cycles_per_warp_main_5_join_build_6, -1, sizeof(int64_t) * main_5_join_build_6_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 1);
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)region_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_6, d_BUF_IDX_6, d_HT_6.ref(cuco::insert), d_cycles_per_warp_main_5_join_build_6, d_cycles_per_warp_main_5_selection_4, d_region__r_name, d_region__r_regionkey, region_size);
int64_t* cycles_per_warp_main_5_selection_4 = (int64_t*)malloc(sizeof(int64_t) * main_5_selection_4_cpw_size);
hipMemcpy(cycles_per_warp_main_5_selection_4, d_cycles_per_warp_main_5_selection_4, sizeof(int64_t) * main_5_selection_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_selection_4 ";
for (auto i=0ull; i < main_5_selection_4_cpw_size; i++) std::cout << cycles_per_warp_main_5_selection_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_join_build_6 = (int64_t*)malloc(sizeof(int64_t) * main_5_join_build_6_cpw_size);
hipMemcpy(cycles_per_warp_main_5_join_build_6, d_cycles_per_warp_main_5_join_build_6, sizeof(int64_t) * main_5_join_build_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_join_build_6 ";
for (auto i=0ull; i < main_5_join_build_6_cpw_size; i++) std::cout << cycles_per_warp_main_5_join_build_6[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_9_selection_8;
auto main_9_selection_8_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_9_selection_8, sizeof(int64_t) * main_9_selection_8_cpw_size);
hipMemset(d_cycles_per_warp_main_9_selection_8, -1, sizeof(int64_t) * main_9_selection_8_cpw_size);
//Materialize count
uint64_t* d_COUNT10;
hipMalloc(&d_COUNT10, sizeof(uint64_t));
hipMemset(d_COUNT10, 0, sizeof(uint64_t));
count_9<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT10, d_orders__o_orderdate, orders_size);
uint64_t COUNT10;
hipMemcpy(&COUNT10, d_COUNT10, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_9_join_build_10;
auto main_9_join_build_10_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_9_join_build_10, sizeof(int64_t) * main_9_join_build_10_cpw_size);
hipMemset(d_cycles_per_warp_main_9_join_build_10, -1, sizeof(int64_t) * main_9_join_build_10_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_10;
hipMalloc(&d_BUF_IDX_10, sizeof(uint64_t));
hipMemset(d_BUF_IDX_10, 0, sizeof(uint64_t));
uint64_t* d_BUF_10;
hipMalloc(&d_BUF_10, sizeof(uint64_t) * COUNT10 * 1);
auto d_HT_10 = cuco::static_map{ (int)COUNT10*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_9<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_10, d_BUF_IDX_10, d_HT_10.ref(cuco::insert), d_cycles_per_warp_main_9_join_build_10, d_cycles_per_warp_main_9_selection_8, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
int64_t* cycles_per_warp_main_9_selection_8 = (int64_t*)malloc(sizeof(int64_t) * main_9_selection_8_cpw_size);
hipMemcpy(cycles_per_warp_main_9_selection_8, d_cycles_per_warp_main_9_selection_8, sizeof(int64_t) * main_9_selection_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_9_selection_8 ";
for (auto i=0ull; i < main_9_selection_8_cpw_size; i++) std::cout << cycles_per_warp_main_9_selection_8[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_9_join_build_10 = (int64_t*)malloc(sizeof(int64_t) * main_9_join_build_10_cpw_size);
hipMemcpy(cycles_per_warp_main_9_join_build_10, d_cycles_per_warp_main_9_join_build_10, sizeof(int64_t) * main_9_join_build_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_9_join_build_10 ";
for (auto i=0ull; i < main_9_join_build_10_cpw_size; i++) std::cout << cycles_per_warp_main_9_join_build_10[i] << " ";
std::cout << std::endl;
//Materialize count
uint64_t* d_COUNT11;
hipMalloc(&d_COUNT11, sizeof(uint64_t));
hipMemset(d_COUNT11, 0, sizeof(uint64_t));
count_12<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT11, customer_size);
uint64_t COUNT11;
hipMemcpy(&COUNT11, d_COUNT11, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_12_join_build_11;
auto main_12_join_build_11_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_12_join_build_11, sizeof(int64_t) * main_12_join_build_11_cpw_size);
hipMemset(d_cycles_per_warp_main_12_join_build_11, -1, sizeof(int64_t) * main_12_join_build_11_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_11;
hipMalloc(&d_BUF_IDX_11, sizeof(uint64_t));
hipMemset(d_BUF_IDX_11, 0, sizeof(uint64_t));
uint64_t* d_BUF_11;
hipMalloc(&d_BUF_11, sizeof(uint64_t) * COUNT11 * 1);
auto d_HT_11 = cuco::static_map{ (int)COUNT11*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_12<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_11, d_BUF_IDX_11, d_HT_11.ref(cuco::insert), d_customer__c_custkey, customer_size, d_cycles_per_warp_main_12_join_build_11);
int64_t* cycles_per_warp_main_12_join_build_11 = (int64_t*)malloc(sizeof(int64_t) * main_12_join_build_11_cpw_size);
hipMemcpy(cycles_per_warp_main_12_join_build_11, d_cycles_per_warp_main_12_join_build_11, sizeof(int64_t) * main_12_join_build_11_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_12_join_build_11 ";
for (auto i=0ull; i < main_12_join_build_11_cpw_size; i++) std::cout << cycles_per_warp_main_12_join_build_11[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_7_join_probe_6;
auto main_7_join_probe_6_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_probe_6, sizeof(int64_t) * main_7_join_probe_6_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_probe_6, -1, sizeof(int64_t) * main_7_join_probe_6_cpw_size);
//Materialize count
uint64_t* d_COUNT13;
hipMalloc(&d_COUNT13, sizeof(uint64_t));
hipMemset(d_COUNT13, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_6, d_COUNT13, d_HT_6.ref(cuco::find), d_nation__n_regionkey, nation_size);
uint64_t COUNT13;
hipMemcpy(&COUNT13, d_COUNT13, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_7_join_build_13;
auto main_7_join_build_13_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_build_13, sizeof(int64_t) * main_7_join_build_13_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_build_13, -1, sizeof(int64_t) * main_7_join_build_13_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_13;
hipMalloc(&d_BUF_IDX_13, sizeof(uint64_t));
hipMemset(d_BUF_IDX_13, 0, sizeof(uint64_t));
uint64_t* d_BUF_13;
hipMalloc(&d_BUF_13, sizeof(uint64_t) * COUNT13 * 2);
auto d_HT_13 = cuco::static_map{ (int)COUNT13*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_13, d_BUF_6, d_BUF_IDX_13, d_HT_13.ref(cuco::insert), d_HT_6.ref(cuco::find), d_cycles_per_warp_main_7_join_build_13, d_cycles_per_warp_main_7_join_probe_6, d_nation__n_nationkey, d_nation__n_regionkey, nation_size);
int64_t* cycles_per_warp_main_7_join_probe_6 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_probe_6_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_probe_6, d_cycles_per_warp_main_7_join_probe_6, sizeof(int64_t) * main_7_join_probe_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_probe_6 ";
for (auto i=0ull; i < main_7_join_probe_6_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_probe_6[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_join_build_13 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_build_13_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_build_13, d_cycles_per_warp_main_7_join_build_13, sizeof(int64_t) * main_7_join_build_13_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_build_13 ";
for (auto i=0ull; i < main_7_join_build_13_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_build_13[i] << " ";
std::cout << std::endl;
//Materialize count
uint64_t* d_COUNT14;
hipMalloc(&d_COUNT14, sizeof(uint64_t));
hipMemset(d_COUNT14, 0, sizeof(uint64_t));
count_15<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT14, supplier_size);
uint64_t COUNT14;
hipMemcpy(&COUNT14, d_COUNT14, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_15_join_build_14;
auto main_15_join_build_14_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_15_join_build_14, sizeof(int64_t) * main_15_join_build_14_cpw_size);
hipMemset(d_cycles_per_warp_main_15_join_build_14, -1, sizeof(int64_t) * main_15_join_build_14_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_14;
hipMalloc(&d_BUF_IDX_14, sizeof(uint64_t));
hipMemset(d_BUF_IDX_14, 0, sizeof(uint64_t));
uint64_t* d_BUF_14;
hipMalloc(&d_BUF_14, sizeof(uint64_t) * COUNT14 * 1);
auto d_HT_14 = cuco::static_map{ (int)COUNT14*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_15<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_14, d_BUF_IDX_14, d_HT_14.ref(cuco::insert), d_cycles_per_warp_main_15_join_build_14, d_supplier__s_suppkey, supplier_size);
int64_t* cycles_per_warp_main_15_join_build_14 = (int64_t*)malloc(sizeof(int64_t) * main_15_join_build_14_cpw_size);
hipMemcpy(cycles_per_warp_main_15_join_build_14, d_cycles_per_warp_main_15_join_build_14, sizeof(int64_t) * main_15_join_build_14_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_15_join_build_14 ";
for (auto i=0ull; i < main_15_join_build_14_cpw_size; i++) std::cout << cycles_per_warp_main_15_join_build_14[i] << " ";
std::cout << std::endl;
//Materialize count
uint64_t* d_COUNT16;
hipMalloc(&d_COUNT16, sizeof(uint64_t));
hipMemset(d_COUNT16, 0, sizeof(uint64_t));
count_17<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT16, nation_size);
uint64_t COUNT16;
hipMemcpy(&COUNT16, d_COUNT16, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_17_join_build_16;
auto main_17_join_build_16_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_17_join_build_16, sizeof(int64_t) * main_17_join_build_16_cpw_size);
hipMemset(d_cycles_per_warp_main_17_join_build_16, -1, sizeof(int64_t) * main_17_join_build_16_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_16;
hipMalloc(&d_BUF_IDX_16, sizeof(uint64_t));
hipMemset(d_BUF_IDX_16, 0, sizeof(uint64_t));
uint64_t* d_BUF_16;
hipMalloc(&d_BUF_16, sizeof(uint64_t) * COUNT16 * 1);
auto d_HT_16 = cuco::static_map{ (int)COUNT16*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_17<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_16, d_BUF_IDX_16, d_HT_16.ref(cuco::insert), d_cycles_per_warp_main_17_join_build_16, d_nation__n_nationkey, nation_size);
int64_t* cycles_per_warp_main_17_join_build_16 = (int64_t*)malloc(sizeof(int64_t) * main_17_join_build_16_cpw_size);
hipMemcpy(cycles_per_warp_main_17_join_build_16, d_cycles_per_warp_main_17_join_build_16, sizeof(int64_t) * main_17_join_build_16_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_17_join_build_16 ";
for (auto i=0ull; i < main_17_join_build_16_cpw_size; i++) std::cout << cycles_per_warp_main_17_join_build_16[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_3_join_probe_2;
auto main_3_join_probe_2_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_probe_2, sizeof(int64_t) * main_3_join_probe_2_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_probe_2, -1, sizeof(int64_t) * main_3_join_probe_2_cpw_size);
int64_t* d_cycles_per_warp_main_3_join_probe_10;
auto main_3_join_probe_10_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_probe_10, sizeof(int64_t) * main_3_join_probe_10_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_probe_10, -1, sizeof(int64_t) * main_3_join_probe_10_cpw_size);
int64_t* d_cycles_per_warp_main_3_join_probe_11;
auto main_3_join_probe_11_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_probe_11, sizeof(int64_t) * main_3_join_probe_11_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_probe_11, -1, sizeof(int64_t) * main_3_join_probe_11_cpw_size);
int64_t* d_cycles_per_warp_main_3_join_probe_13;
auto main_3_join_probe_13_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_probe_13, sizeof(int64_t) * main_3_join_probe_13_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_probe_13, -1, sizeof(int64_t) * main_3_join_probe_13_cpw_size);
int64_t* d_cycles_per_warp_main_3_join_probe_14;
auto main_3_join_probe_14_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_probe_14, sizeof(int64_t) * main_3_join_probe_14_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_probe_14, -1, sizeof(int64_t) * main_3_join_probe_14_cpw_size);
int64_t* d_cycles_per_warp_main_3_join_probe_16;
auto main_3_join_probe_16_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_probe_16, sizeof(int64_t) * main_3_join_probe_16_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_probe_16, -1, sizeof(int64_t) * main_3_join_probe_16_cpw_size);
int64_t* d_cycles_per_warp_main_3_map_18;
auto main_3_map_18_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_map_18, sizeof(int64_t) * main_3_map_18_cpw_size);
hipMemset(d_cycles_per_warp_main_3_map_18, -1, sizeof(int64_t) * main_3_map_18_cpw_size);
int64_t* d_cycles_per_warp_main_3_map_19;
auto main_3_map_19_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_map_19, sizeof(int64_t) * main_3_map_19_cpw_size);
hipMemset(d_cycles_per_warp_main_3_map_19, -1, sizeof(int64_t) * main_3_map_19_cpw_size);
int64_t* d_cycles_per_warp_main_3_map_20;
auto main_3_map_20_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_map_20, sizeof(int64_t) * main_3_map_20_cpw_size);
hipMemset(d_cycles_per_warp_main_3_map_20, -1, sizeof(int64_t) * main_3_map_20_cpw_size);
//Create aggregation hash table
auto d_HT_21 = cuco::static_map{ (int)5117*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_3<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_10, d_BUF_11, d_BUF_13, d_BUF_14, d_BUF_16, d_BUF_2, d_HT_10.ref(cuco::find), d_HT_11.ref(cuco::find), d_HT_13.ref(cuco::find), d_HT_14.ref(cuco::find), d_HT_16.ref(cuco::find), d_HT_2.ref(cuco::find), d_HT_21.ref(cuco::insert), d_customer__c_nationkey, d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_suppkey, lineitem_size, d_orders__o_custkey, d_orders__o_orderdate, d_supplier__s_nationkey);
size_t COUNT21 = d_HT_21.size();
thrust::device_vector<int64_t> keys_21(COUNT21), vals_21(COUNT21);
d_HT_21.retrieve_all(keys_21.begin(), vals_21.begin());
d_HT_21.clear();
int64_t* raw_keys21 = thrust::raw_pointer_cast(keys_21.data());
insertKeys<<<std::ceil((float)COUNT21/128.), 128>>>(raw_keys21, d_HT_21.ref(cuco::insert), COUNT21);
int64_t* d_cycles_per_warp_main_3_aggregation_21;
auto main_3_aggregation_21_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_aggregation_21, sizeof(int64_t) * main_3_aggregation_21_cpw_size);
hipMemset(d_cycles_per_warp_main_3_aggregation_21, -1, sizeof(int64_t) * main_3_aggregation_21_cpw_size);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr4;
hipMalloc(&d_aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT21);
hipMemset(d_aggr0__tmp_attr4, 0, sizeof(DBDecimalType) * COUNT21);
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT21);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT21);
DBI64Type* d_KEY_21map0__tmp_attr0;
hipMalloc(&d_KEY_21map0__tmp_attr0, sizeof(DBI64Type) * COUNT21);
hipMemset(d_KEY_21map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT21);
main_3<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_10, d_BUF_11, d_BUF_13, d_BUF_14, d_BUF_16, d_BUF_2, d_HT_10.ref(cuco::find), d_HT_11.ref(cuco::find), d_HT_13.ref(cuco::find), d_HT_14.ref(cuco::find), d_HT_16.ref(cuco::find), d_HT_2.ref(cuco::find), d_HT_21.ref(cuco::find), d_KEY_21map0__tmp_attr0, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_customer__c_nationkey, d_cycles_per_warp_main_3_aggregation_21, d_cycles_per_warp_main_3_join_probe_10, d_cycles_per_warp_main_3_join_probe_11, d_cycles_per_warp_main_3_join_probe_13, d_cycles_per_warp_main_3_join_probe_14, d_cycles_per_warp_main_3_join_probe_16, d_cycles_per_warp_main_3_join_probe_2, d_cycles_per_warp_main_3_map_18, d_cycles_per_warp_main_3_map_19, d_cycles_per_warp_main_3_map_20, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_orders__o_custkey, d_orders__o_orderdate, d_supplier__s_nationkey);
int64_t* cycles_per_warp_main_3_join_probe_2 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_probe_2_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_probe_2, d_cycles_per_warp_main_3_join_probe_2, sizeof(int64_t) * main_3_join_probe_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_probe_2 ";
for (auto i=0ull; i < main_3_join_probe_2_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_probe_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_join_probe_10 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_probe_10_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_probe_10, d_cycles_per_warp_main_3_join_probe_10, sizeof(int64_t) * main_3_join_probe_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_probe_10 ";
for (auto i=0ull; i < main_3_join_probe_10_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_probe_10[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_join_probe_11 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_probe_11_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_probe_11, d_cycles_per_warp_main_3_join_probe_11, sizeof(int64_t) * main_3_join_probe_11_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_probe_11 ";
for (auto i=0ull; i < main_3_join_probe_11_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_probe_11[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_join_probe_13 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_probe_13_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_probe_13, d_cycles_per_warp_main_3_join_probe_13, sizeof(int64_t) * main_3_join_probe_13_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_probe_13 ";
for (auto i=0ull; i < main_3_join_probe_13_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_probe_13[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_join_probe_14 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_probe_14_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_probe_14, d_cycles_per_warp_main_3_join_probe_14, sizeof(int64_t) * main_3_join_probe_14_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_probe_14 ";
for (auto i=0ull; i < main_3_join_probe_14_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_probe_14[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_join_probe_16 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_probe_16_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_probe_16, d_cycles_per_warp_main_3_join_probe_16, sizeof(int64_t) * main_3_join_probe_16_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_probe_16 ";
for (auto i=0ull; i < main_3_join_probe_16_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_probe_16[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_map_18 = (int64_t*)malloc(sizeof(int64_t) * main_3_map_18_cpw_size);
hipMemcpy(cycles_per_warp_main_3_map_18, d_cycles_per_warp_main_3_map_18, sizeof(int64_t) * main_3_map_18_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_map_18 ";
for (auto i=0ull; i < main_3_map_18_cpw_size; i++) std::cout << cycles_per_warp_main_3_map_18[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_map_19 = (int64_t*)malloc(sizeof(int64_t) * main_3_map_19_cpw_size);
hipMemcpy(cycles_per_warp_main_3_map_19, d_cycles_per_warp_main_3_map_19, sizeof(int64_t) * main_3_map_19_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_map_19 ";
for (auto i=0ull; i < main_3_map_19_cpw_size; i++) std::cout << cycles_per_warp_main_3_map_19[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_map_20 = (int64_t*)malloc(sizeof(int64_t) * main_3_map_20_cpw_size);
hipMemcpy(cycles_per_warp_main_3_map_20, d_cycles_per_warp_main_3_map_20, sizeof(int64_t) * main_3_map_20_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_map_20 ";
for (auto i=0ull; i < main_3_map_20_cpw_size; i++) std::cout << cycles_per_warp_main_3_map_20[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_aggregation_21 = (int64_t*)malloc(sizeof(int64_t) * main_3_aggregation_21_cpw_size);
hipMemcpy(cycles_per_warp_main_3_aggregation_21, d_cycles_per_warp_main_3_aggregation_21, sizeof(int64_t) * main_3_aggregation_21_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_aggregation_21 ";
for (auto i=0ull; i < main_3_aggregation_21_cpw_size; i++) std::cout << cycles_per_warp_main_3_aggregation_21[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_23_map_22;
auto main_23_map_22_cpw_size = std::ceil((float)COUNT21/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_23_map_22, sizeof(int64_t) * main_23_map_22_cpw_size);
hipMemset(d_cycles_per_warp_main_23_map_22, -1, sizeof(int64_t) * main_23_map_22_cpw_size);
//Materialize count
uint64_t* d_COUNT24;
hipMalloc(&d_COUNT24, sizeof(uint64_t));
hipMemset(d_COUNT24, 0, sizeof(uint64_t));
count_23<<<std::ceil((float)COUNT21/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT21, d_COUNT24);
uint64_t COUNT24;
hipMemcpy(&COUNT24, d_COUNT24, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_23_materialize_24;
auto main_23_materialize_24_cpw_size = std::ceil((float)COUNT21/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_23_materialize_24, sizeof(int64_t) * main_23_materialize_24_cpw_size);
hipMemset(d_cycles_per_warp_main_23_materialize_24, -1, sizeof(int64_t) * main_23_materialize_24_cpw_size);
//Materialize buffers
uint64_t* d_MAT_IDX24;
hipMalloc(&d_MAT_IDX24, sizeof(uint64_t));
hipMemset(d_MAT_IDX24, 0, sizeof(uint64_t));
auto MAT24map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT24);
DBI64Type* d_MAT24map0__tmp_attr0;
hipMalloc(&d_MAT24map0__tmp_attr0, sizeof(DBI64Type) * COUNT24);
auto MAT24map2__tmp_attr5 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT24);
DBDecimalType* d_MAT24map2__tmp_attr5;
hipMalloc(&d_MAT24map2__tmp_attr5, sizeof(DBDecimalType) * COUNT24);
main_23<<<std::ceil((float)COUNT21/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT21, d_MAT24map0__tmp_attr0, d_MAT24map2__tmp_attr5, d_MAT_IDX24, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_cycles_per_warp_main_23_map_22, d_cycles_per_warp_main_23_materialize_24, d_KEY_21map0__tmp_attr0);
hipMemcpy(MAT24map0__tmp_attr0, d_MAT24map0__tmp_attr0, sizeof(DBI64Type) * COUNT24, hipMemcpyDeviceToHost);
hipMemcpy(MAT24map2__tmp_attr5, d_MAT24map2__tmp_attr5, sizeof(DBDecimalType) * COUNT24, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_23_map_22 = (int64_t*)malloc(sizeof(int64_t) * main_23_map_22_cpw_size);
hipMemcpy(cycles_per_warp_main_23_map_22, d_cycles_per_warp_main_23_map_22, sizeof(int64_t) * main_23_map_22_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_23_map_22 ";
for (auto i=0ull; i < main_23_map_22_cpw_size; i++) std::cout << cycles_per_warp_main_23_map_22[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_23_materialize_24 = (int64_t*)malloc(sizeof(int64_t) * main_23_materialize_24_cpw_size);
hipMemcpy(cycles_per_warp_main_23_materialize_24, d_cycles_per_warp_main_23_materialize_24, sizeof(int64_t) * main_23_materialize_24_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_23_materialize_24 ";
for (auto i=0ull; i < main_23_materialize_24_cpw_size; i++) std::cout << cycles_per_warp_main_23_materialize_24[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
      size_t aux_mem = usedGpuMem() - used_mem;
      std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_COUNT2);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_COUNT6);
hipFree(d_BUF_10);
hipFree(d_BUF_IDX_10);
hipFree(d_COUNT10);
hipFree(d_BUF_11);
hipFree(d_BUF_IDX_11);
hipFree(d_COUNT11);
hipFree(d_BUF_13);
hipFree(d_BUF_IDX_13);
hipFree(d_COUNT13);
hipFree(d_BUF_14);
hipFree(d_BUF_IDX_14);
hipFree(d_COUNT14);
hipFree(d_BUF_16);
hipFree(d_BUF_IDX_16);
hipFree(d_COUNT16);
hipFree(d_KEY_21map0__tmp_attr0);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_aggr0__tmp_attr4);
hipFree(d_COUNT24);
hipFree(d_MAT24map0__tmp_attr0);
hipFree(d_MAT24map2__tmp_attr5);
hipFree(d_MAT_IDX24);
free(MAT24map0__tmp_attr0);
free(MAT24map2__tmp_attr5);
}