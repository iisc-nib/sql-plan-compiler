#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_64352c889080(uint64_t* COUNT64352c878750, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT64352c878750, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_64352c889080(uint64_t* BUF_64352c878750, uint64_t* BUF_IDX_64352c878750, HASHTABLE_INSERT HT_64352c878750, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_64352c878750 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_64352c878750 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_64352c878750 = atomicAdd((int*)BUF_IDX_64352c878750, 1);
HT_64352c878750.insert(cuco::pair{KEY_64352c878750, buf_idx_64352c878750});
BUF_64352c878750[buf_idx_64352c878750 * 1 + 0] = tid;
}
__global__ void count_64352c8210e0(uint64_t* COUNT64352c87c300, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT64352c87c300, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_64352c8210e0(uint64_t* BUF_64352c87c300, uint64_t* BUF_IDX_64352c87c300, HASHTABLE_INSERT HT_64352c87c300, DBI32Type* customer__c_custkey, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_64352c87c300 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_64352c87c300 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_64352c87c300 = atomicAdd((int*)BUF_IDX_64352c87c300, 1);
HT_64352c87c300.insert(cuco::pair{KEY_64352c87c300, buf_idx_64352c87c300});
BUF_64352c87c300[buf_idx_64352c87c300 * 1 + 0] = tid;
}
__global__ void count_64352c821320(uint64_t* COUNT64352c859860, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT64352c859860, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_64352c821320(uint64_t* BUF_64352c859860, uint64_t* BUF_IDX_64352c859860, HASHTABLE_INSERT HT_64352c859860, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!((evaluatePredicate(reg_date__d_year, 1997, Predicate::eq)) || (evaluatePredicate(reg_date__d_year, 1998, Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_64352c859860 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_64352c859860 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_64352c859860 = atomicAdd((int*)BUF_IDX_64352c859860, 1);
HT_64352c859860.insert(cuco::pair{KEY_64352c859860, buf_idx_64352c859860});
BUF_64352c859860[buf_idx_64352c859860 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_64352c88e100(uint64_t* BUF_64352c859860, uint64_t* BUF_64352c878750, uint64_t* BUF_64352c87c300, uint64_t* COUNT64352c876900, HASHTABLE_PROBE HT_64352c859860, HASHTABLE_PROBE HT_64352c878750, HASHTABLE_PROBE HT_64352c87c300, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_64352c878750 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_64352c878750 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_64352c878750.for_each(KEY_64352c878750, [&] __device__ (auto const SLOT_64352c878750) {

auto const [slot_first64352c878750, slot_second64352c878750] = SLOT_64352c878750;
if (!(true)) return;
uint64_t KEY_64352c87c300 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_64352c87c300 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_64352c87c300.for_each(KEY_64352c87c300, [&] __device__ (auto const SLOT_64352c87c300) {

auto const [slot_first64352c87c300, slot_second64352c87c300] = SLOT_64352c87c300;
if (!(true)) return;
uint64_t KEY_64352c859860 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_64352c859860 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_64352c859860.for_each(KEY_64352c859860, [&] __device__ (auto const SLOT_64352c859860) {

auto const [slot_first64352c859860, slot_second64352c859860] = SLOT_64352c859860;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT64352c876900, 1);
});
});
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_64352c88e100(uint64_t* BUF_64352c859860, uint64_t* BUF_64352c876900, uint64_t* BUF_64352c878750, uint64_t* BUF_64352c87c300, uint64_t* BUF_IDX_64352c876900, HASHTABLE_PROBE HT_64352c859860, HASHTABLE_INSERT HT_64352c876900, HASHTABLE_PROBE HT_64352c878750, HASHTABLE_PROBE HT_64352c87c300, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_64352c878750 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_64352c878750 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_64352c878750.for_each(KEY_64352c878750, [&] __device__ (auto const SLOT_64352c878750) {
auto const [slot_first64352c878750, slot_second64352c878750] = SLOT_64352c878750;
if (!(true)) return;
uint64_t KEY_64352c87c300 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_64352c87c300 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_64352c87c300.for_each(KEY_64352c87c300, [&] __device__ (auto const SLOT_64352c87c300) {
auto const [slot_first64352c87c300, slot_second64352c87c300] = SLOT_64352c87c300;
if (!(true)) return;
uint64_t KEY_64352c859860 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_64352c859860 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_64352c859860.for_each(KEY_64352c859860, [&] __device__ (auto const SLOT_64352c859860) {
auto const [slot_first64352c859860, slot_second64352c859860] = SLOT_64352c859860;
if (!(true)) return;
uint64_t KEY_64352c876900 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_64352c876900 |= reg_lineorder__lo_partkey;
// Insert hash table kernel;
auto buf_idx_64352c876900 = atomicAdd((int*)BUF_IDX_64352c876900, 1);
HT_64352c876900.insert(cuco::pair{KEY_64352c876900, buf_idx_64352c876900});
BUF_64352c876900[buf_idx_64352c876900 * 4 + 0] = BUF_64352c859860[slot_second64352c859860 * 1 + 0];
BUF_64352c876900[buf_idx_64352c876900 * 4 + 1] = BUF_64352c87c300[slot_second64352c87c300 * 1 + 0];
BUF_64352c876900[buf_idx_64352c876900 * 4 + 2] = BUF_64352c878750[slot_second64352c878750 * 1 + 0];
BUF_64352c876900[buf_idx_64352c876900 * 4 + 3] = tid;
});
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_64352c88b8a0(uint64_t* BUF_64352c876900, HASHTABLE_INSERT HT_64352c82ff40, HASHTABLE_PROBE HT_64352c876900, DBI32Type* date__d_year, DBI16Type* part__p_category_encoded, DBStringType* part__p_mfgr, DBI32Type* part__p_partkey, size_t part_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_mfgr = part__p_mfgr[tid];
if (!((evaluatePredicate(reg_part__p_mfgr, "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr, "MFGR#2", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_64352c876900 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_64352c876900 |= reg_part__p_partkey;
//Probe Hash table
HT_64352c876900.for_each(KEY_64352c876900, [&] __device__ (auto const SLOT_64352c876900) {

auto const [slot_first64352c876900, slot_second64352c876900] = SLOT_64352c876900;
if (!(true)) return;
uint64_t KEY_64352c82ff40 = 0;
auto reg_date__d_year = date__d_year[BUF_64352c876900[slot_second64352c876900 * 4 + 0]];

KEY_64352c82ff40 |= reg_date__d_year;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_64352c876900[slot_second64352c876900 * 4 + 2]];
KEY_64352c82ff40 <<= 16;
KEY_64352c82ff40 |= reg_supplier__s_nation_encoded;
auto reg_part__p_category_encoded = part__p_category_encoded[tid];
KEY_64352c82ff40 <<= 16;
KEY_64352c82ff40 |= reg_part__p_category_encoded;
//Create aggregation hash table
HT_64352c82ff40.insert(cuco::pair{KEY_64352c82ff40, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_64352c88b8a0(uint64_t* BUF_64352c876900, HASHTABLE_FIND HT_64352c82ff40, HASHTABLE_PROBE HT_64352c876900, DBI32Type* KEY_64352c82ff40date__d_year, DBI16Type* KEY_64352c82ff40part__p_category_encoded, DBI16Type* KEY_64352c82ff40supplier__s_nation_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBDecimalType* lineorder__lo_revenue, DBDecimalType* lineorder__lo_supplycost, DBI16Type* part__p_category_encoded, DBStringType* part__p_mfgr, DBI32Type* part__p_partkey, size_t part_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_mfgr = part__p_mfgr[tid];
if (!((evaluatePredicate(reg_part__p_mfgr, "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr, "MFGR#2", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_64352c876900 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_64352c876900 |= reg_part__p_partkey;
//Probe Hash table
HT_64352c876900.for_each(KEY_64352c876900, [&] __device__ (auto const SLOT_64352c876900) {
auto const [slot_first64352c876900, slot_second64352c876900] = SLOT_64352c876900;
if (!(true)) return;
uint64_t KEY_64352c82ff40 = 0;
auto reg_date__d_year = date__d_year[BUF_64352c876900[slot_second64352c876900 * 4 + 0]];

KEY_64352c82ff40 |= reg_date__d_year;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_64352c876900[slot_second64352c876900 * 4 + 2]];
KEY_64352c82ff40 <<= 16;
KEY_64352c82ff40 |= reg_supplier__s_nation_encoded;
auto reg_part__p_category_encoded = part__p_category_encoded[tid];
KEY_64352c82ff40 <<= 16;
KEY_64352c82ff40 |= reg_part__p_category_encoded;
//Aggregate in hashtable
auto buf_idx_64352c82ff40 = HT_64352c82ff40.find(KEY_64352c82ff40)->second;
auto reg_lineorder__lo_supplycost = lineorder__lo_supplycost[BUF_64352c876900[slot_second64352c876900 * 4 + 3]];
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_64352c876900[slot_second64352c876900 * 4 + 3]];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_revenue) - (reg_lineorder__lo_supplycost);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_64352c82ff40], reg_map0__tmp_attr1);
KEY_64352c82ff40date__d_year[buf_idx_64352c82ff40] = reg_date__d_year;
KEY_64352c82ff40supplier__s_nation_encoded[buf_idx_64352c82ff40] = reg_supplier__s_nation_encoded;
KEY_64352c82ff40part__p_category_encoded[buf_idx_64352c82ff40] = reg_part__p_category_encoded;
});
}
__global__ void count_64352c89e1d0(uint64_t* COUNT64352c80fc50, size_t COUNT64352c82ff40) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT64352c82ff40) return;
//Materialize count
atomicAdd((int*)COUNT64352c80fc50, 1);
}
__global__ void main_64352c89e1d0(size_t COUNT64352c82ff40, DBDecimalType* MAT64352c80fc50aggr0__tmp_attr0, DBI32Type* MAT64352c80fc50date__d_year, DBI16Type* MAT64352c80fc50part__p_category_encoded, DBI16Type* MAT64352c80fc50supplier__s_nation_encoded, uint64_t* MAT_IDX64352c80fc50, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_category_encoded, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT64352c82ff40) return;
//Materialize buffers
auto mat_idx64352c80fc50 = atomicAdd((int*)MAT_IDX64352c80fc50, 1);
auto reg_date__d_year = date__d_year[tid];
MAT64352c80fc50date__d_year[mat_idx64352c80fc50] = reg_date__d_year;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[tid];
MAT64352c80fc50supplier__s_nation_encoded[mat_idx64352c80fc50] = reg_supplier__s_nation_encoded;
auto reg_part__p_category_encoded = part__p_category_encoded[tid];
MAT64352c80fc50part__p_category_encoded[mat_idx64352c80fc50] = reg_part__p_category_encoded;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT64352c80fc50aggr0__tmp_attr0[mat_idx64352c80fc50] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT64352c878750;
hipMalloc(&d_COUNT64352c878750, sizeof(uint64_t));
hipMemset(d_COUNT64352c878750, 0, sizeof(uint64_t));
count_64352c889080<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT64352c878750, d_supplier__s_region, supplier_size);
uint64_t COUNT64352c878750;
hipMemcpy(&COUNT64352c878750, d_COUNT64352c878750, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_64352c878750;
hipMalloc(&d_BUF_IDX_64352c878750, sizeof(uint64_t));
hipMemset(d_BUF_IDX_64352c878750, 0, sizeof(uint64_t));
uint64_t* d_BUF_64352c878750;
hipMalloc(&d_BUF_64352c878750, sizeof(uint64_t) * COUNT64352c878750 * 1);
auto d_HT_64352c878750 = cuco::experimental::static_multimap{ (int)COUNT64352c878750*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_64352c889080<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_64352c878750, d_BUF_IDX_64352c878750, d_HT_64352c878750.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT64352c87c300;
hipMalloc(&d_COUNT64352c87c300, sizeof(uint64_t));
hipMemset(d_COUNT64352c87c300, 0, sizeof(uint64_t));
count_64352c8210e0<<<std::ceil((float)customer_size/128.), 128>>>(d_COUNT64352c87c300, d_customer__c_region, customer_size);
uint64_t COUNT64352c87c300;
hipMemcpy(&COUNT64352c87c300, d_COUNT64352c87c300, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_64352c87c300;
hipMalloc(&d_BUF_IDX_64352c87c300, sizeof(uint64_t));
hipMemset(d_BUF_IDX_64352c87c300, 0, sizeof(uint64_t));
uint64_t* d_BUF_64352c87c300;
hipMalloc(&d_BUF_64352c87c300, sizeof(uint64_t) * COUNT64352c87c300 * 1);
auto d_HT_64352c87c300 = cuco::experimental::static_multimap{ (int)COUNT64352c87c300*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_64352c8210e0<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_64352c87c300, d_BUF_IDX_64352c87c300, d_HT_64352c87c300.ref(cuco::insert), d_customer__c_custkey, d_customer__c_region, customer_size);
//Materialize count
uint64_t* d_COUNT64352c859860;
hipMalloc(&d_COUNT64352c859860, sizeof(uint64_t));
hipMemset(d_COUNT64352c859860, 0, sizeof(uint64_t));
count_64352c821320<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT64352c859860, d_date__d_year, date_size);
uint64_t COUNT64352c859860;
hipMemcpy(&COUNT64352c859860, d_COUNT64352c859860, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_64352c859860;
hipMalloc(&d_BUF_IDX_64352c859860, sizeof(uint64_t));
hipMemset(d_BUF_IDX_64352c859860, 0, sizeof(uint64_t));
uint64_t* d_BUF_64352c859860;
hipMalloc(&d_BUF_64352c859860, sizeof(uint64_t) * COUNT64352c859860 * 1);
auto d_HT_64352c859860 = cuco::experimental::static_multimap{ (int)COUNT64352c859860*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_64352c821320<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_64352c859860, d_BUF_IDX_64352c859860, d_HT_64352c859860.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Materialize count
uint64_t* d_COUNT64352c876900;
hipMalloc(&d_COUNT64352c876900, sizeof(uint64_t));
hipMemset(d_COUNT64352c876900, 0, sizeof(uint64_t));
count_64352c88e100<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_64352c859860, d_BUF_64352c878750, d_BUF_64352c87c300, d_COUNT64352c876900, d_HT_64352c859860.ref(cuco::for_each), d_HT_64352c878750.ref(cuco::for_each), d_HT_64352c87c300.ref(cuco::for_each), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size);
uint64_t COUNT64352c876900;
hipMemcpy(&COUNT64352c876900, d_COUNT64352c876900, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_64352c876900;
hipMalloc(&d_BUF_IDX_64352c876900, sizeof(uint64_t));
hipMemset(d_BUF_IDX_64352c876900, 0, sizeof(uint64_t));
uint64_t* d_BUF_64352c876900;
hipMalloc(&d_BUF_64352c876900, sizeof(uint64_t) * COUNT64352c876900 * 4);
auto d_HT_64352c876900 = cuco::experimental::static_multimap{ (int)COUNT64352c876900*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_64352c88e100<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_64352c859860, d_BUF_64352c876900, d_BUF_64352c878750, d_BUF_64352c87c300, d_BUF_IDX_64352c876900, d_HT_64352c859860.ref(cuco::for_each), d_HT_64352c876900.ref(cuco::insert), d_HT_64352c878750.ref(cuco::for_each), d_HT_64352c87c300.ref(cuco::for_each), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
//Create aggregation hash table
auto d_HT_64352c82ff40 = cuco::static_map{ (int)24650*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_64352c88b8a0<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_64352c876900, d_HT_64352c82ff40.ref(cuco::insert), d_HT_64352c876900.ref(cuco::for_each), d_date__d_year, d_part__p_category_encoded, d_part__p_mfgr, d_part__p_partkey, part_size, d_supplier__s_nation_encoded);
size_t COUNT64352c82ff40 = d_HT_64352c82ff40.size();
thrust::device_vector<int64_t> keys_64352c82ff40(COUNT64352c82ff40), vals_64352c82ff40(COUNT64352c82ff40);
d_HT_64352c82ff40.retrieve_all(keys_64352c82ff40.begin(), vals_64352c82ff40.begin());
d_HT_64352c82ff40.clear();
int64_t* raw_keys64352c82ff40 = thrust::raw_pointer_cast(keys_64352c82ff40.data());
insertKeys<<<std::ceil((float)COUNT64352c82ff40/128.), 128>>>(raw_keys64352c82ff40, d_HT_64352c82ff40.ref(cuco::insert), COUNT64352c82ff40);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT64352c82ff40);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT64352c82ff40);
DBI32Type* d_KEY_64352c82ff40date__d_year;
hipMalloc(&d_KEY_64352c82ff40date__d_year, sizeof(DBI32Type) * COUNT64352c82ff40);
hipMemset(d_KEY_64352c82ff40date__d_year, 0, sizeof(DBI32Type) * COUNT64352c82ff40);
DBI16Type* d_KEY_64352c82ff40supplier__s_nation_encoded;
hipMalloc(&d_KEY_64352c82ff40supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT64352c82ff40);
hipMemset(d_KEY_64352c82ff40supplier__s_nation_encoded, 0, sizeof(DBI16Type) * COUNT64352c82ff40);
DBI16Type* d_KEY_64352c82ff40part__p_category_encoded;
hipMalloc(&d_KEY_64352c82ff40part__p_category_encoded, sizeof(DBI16Type) * COUNT64352c82ff40);
hipMemset(d_KEY_64352c82ff40part__p_category_encoded, 0, sizeof(DBI16Type) * COUNT64352c82ff40);
main_64352c88b8a0<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_64352c876900, d_HT_64352c82ff40.ref(cuco::find), d_HT_64352c876900.ref(cuco::for_each), d_KEY_64352c82ff40date__d_year, d_KEY_64352c82ff40part__p_category_encoded, d_KEY_64352c82ff40supplier__s_nation_encoded, d_aggr0__tmp_attr0, d_date__d_year, d_lineorder__lo_revenue, d_lineorder__lo_supplycost, d_part__p_category_encoded, d_part__p_mfgr, d_part__p_partkey, part_size, d_supplier__s_nation_encoded);
//Materialize count
uint64_t* d_COUNT64352c80fc50;
hipMalloc(&d_COUNT64352c80fc50, sizeof(uint64_t));
hipMemset(d_COUNT64352c80fc50, 0, sizeof(uint64_t));
count_64352c89e1d0<<<std::ceil((float)COUNT64352c82ff40/128.), 128>>>(d_COUNT64352c80fc50, COUNT64352c82ff40);
uint64_t COUNT64352c80fc50;
hipMemcpy(&COUNT64352c80fc50, d_COUNT64352c80fc50, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX64352c80fc50;
hipMalloc(&d_MAT_IDX64352c80fc50, sizeof(uint64_t));
hipMemset(d_MAT_IDX64352c80fc50, 0, sizeof(uint64_t));
auto MAT64352c80fc50date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT64352c80fc50);
DBI32Type* d_MAT64352c80fc50date__d_year;
hipMalloc(&d_MAT64352c80fc50date__d_year, sizeof(DBI32Type) * COUNT64352c80fc50);
auto MAT64352c80fc50supplier__s_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT64352c80fc50);
DBI16Type* d_MAT64352c80fc50supplier__s_nation_encoded;
hipMalloc(&d_MAT64352c80fc50supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT64352c80fc50);
auto MAT64352c80fc50part__p_category_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT64352c80fc50);
DBI16Type* d_MAT64352c80fc50part__p_category_encoded;
hipMalloc(&d_MAT64352c80fc50part__p_category_encoded, sizeof(DBI16Type) * COUNT64352c80fc50);
auto MAT64352c80fc50aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT64352c80fc50);
DBDecimalType* d_MAT64352c80fc50aggr0__tmp_attr0;
hipMalloc(&d_MAT64352c80fc50aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT64352c80fc50);
main_64352c89e1d0<<<std::ceil((float)COUNT64352c82ff40/128.), 128>>>(COUNT64352c82ff40, d_MAT64352c80fc50aggr0__tmp_attr0, d_MAT64352c80fc50date__d_year, d_MAT64352c80fc50part__p_category_encoded, d_MAT64352c80fc50supplier__s_nation_encoded, d_MAT_IDX64352c80fc50, d_aggr0__tmp_attr0, d_KEY_64352c82ff40date__d_year, d_KEY_64352c82ff40part__p_category_encoded, d_KEY_64352c82ff40supplier__s_nation_encoded);
hipMemcpy(MAT64352c80fc50date__d_year, d_MAT64352c80fc50date__d_year, sizeof(DBI32Type) * COUNT64352c80fc50, hipMemcpyDeviceToHost);
hipMemcpy(MAT64352c80fc50supplier__s_nation_encoded, d_MAT64352c80fc50supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT64352c80fc50, hipMemcpyDeviceToHost);
hipMemcpy(MAT64352c80fc50part__p_category_encoded, d_MAT64352c80fc50part__p_category_encoded, sizeof(DBI16Type) * COUNT64352c80fc50, hipMemcpyDeviceToHost);
hipMemcpy(MAT64352c80fc50aggr0__tmp_attr0, d_MAT64352c80fc50aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT64352c80fc50, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT64352c80fc50; i++) { std::cout << MAT64352c80fc50date__d_year[i] << "\t";
std::cout << supplier__s_nation_map[MAT64352c80fc50supplier__s_nation_encoded[i]] << "\t";
std::cout << part__p_category_map[MAT64352c80fc50part__p_category_encoded[i]] << "\t";
std::cout << MAT64352c80fc50aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_64352c878750);
hipFree(d_BUF_IDX_64352c878750);
hipFree(d_COUNT64352c878750);
hipFree(d_BUF_64352c87c300);
hipFree(d_BUF_IDX_64352c87c300);
hipFree(d_COUNT64352c87c300);
hipFree(d_BUF_64352c859860);
hipFree(d_BUF_IDX_64352c859860);
hipFree(d_COUNT64352c859860);
hipFree(d_BUF_64352c876900);
hipFree(d_BUF_IDX_64352c876900);
hipFree(d_COUNT64352c876900);
hipFree(d_KEY_64352c82ff40date__d_year);
hipFree(d_KEY_64352c82ff40part__p_category_encoded);
hipFree(d_KEY_64352c82ff40supplier__s_nation_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT64352c80fc50);
hipFree(d_MAT64352c80fc50aggr0__tmp_attr0);
hipFree(d_MAT64352c80fc50date__d_year);
hipFree(d_MAT64352c80fc50part__p_category_encoded);
hipFree(d_MAT64352c80fc50supplier__s_nation_encoded);
hipFree(d_MAT_IDX64352c80fc50);
free(MAT64352c80fc50aggr0__tmp_attr0);
free(MAT64352c80fc50date__d_year);
free(MAT64352c80fc50part__p_category_encoded);
free(MAT64352c80fc50supplier__s_nation_encoded);
}