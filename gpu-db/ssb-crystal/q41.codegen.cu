#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5efabf2f9580(uint64_t* COUNT5efabf332630, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5efabf332630, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5efabf2f9580(uint64_t* BUF_5efabf332630, uint64_t* BUF_IDX_5efabf332630, HASHTABLE_INSERT HT_5efabf332630, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5efabf332630 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5efabf332630 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5efabf332630 = atomicAdd((int*)BUF_IDX_5efabf332630, 1);
HT_5efabf332630.insert(cuco::pair{KEY_5efabf332630, buf_idx_5efabf332630});
BUF_5efabf332630[buf_idx_5efabf332630 * 1 + 0] = tid;
}
__global__ void count_5efabf2f97c0(uint64_t* COUNT5efabf350440, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5efabf350440, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5efabf2f97c0(uint64_t* BUF_5efabf350440, uint64_t* BUF_IDX_5efabf350440, HASHTABLE_INSERT HT_5efabf350440, DBI32Type* customer__c_custkey, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "AMERICA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5efabf350440 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5efabf350440 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_5efabf350440 = atomicAdd((int*)BUF_IDX_5efabf350440, 1);
HT_5efabf350440.insert(cuco::pair{KEY_5efabf350440, buf_idx_5efabf350440});
BUF_5efabf350440[buf_idx_5efabf350440 * 1 + 0] = tid;
}
__global__ void count_5efabf362f80(uint64_t* COUNT5efabf354720, DBStringType* part__p_mfgr, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_mfgr = part__p_mfgr[tid];
if (!((evaluatePredicate(reg_part__p_mfgr, "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr, "MFGR#2", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5efabf354720, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5efabf362f80(uint64_t* BUF_5efabf354720, uint64_t* BUF_IDX_5efabf354720, HASHTABLE_INSERT HT_5efabf354720, DBStringType* part__p_mfgr, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_mfgr = part__p_mfgr[tid];
if (!((evaluatePredicate(reg_part__p_mfgr, "MFGR#1", Predicate::eq)) || (evaluatePredicate(reg_part__p_mfgr, "MFGR#2", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5efabf354720 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_5efabf354720 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_5efabf354720 = atomicAdd((int*)BUF_IDX_5efabf354720, 1);
HT_5efabf354720.insert(cuco::pair{KEY_5efabf354720, buf_idx_5efabf354720});
BUF_5efabf354720[buf_idx_5efabf354720 * 1 + 0] = tid;
}
__global__ void count_5efabf352b40(uint64_t* COUNT5efabf3547e0, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5efabf3547e0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5efabf352b40(uint64_t* BUF_5efabf3547e0, uint64_t* BUF_IDX_5efabf3547e0, HASHTABLE_INSERT HT_5efabf3547e0, DBI32Type* date__d_datekey, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5efabf3547e0 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5efabf3547e0 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_5efabf3547e0 = atomicAdd((int*)BUF_IDX_5efabf3547e0, 1);
HT_5efabf3547e0.insert(cuco::pair{KEY_5efabf3547e0, buf_idx_5efabf3547e0});
BUF_5efabf3547e0[buf_idx_5efabf3547e0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5efabf3657e0(uint64_t* BUF_5efabf332630, uint64_t* BUF_5efabf350440, uint64_t* BUF_5efabf354720, uint64_t* BUF_5efabf3547e0, HASHTABLE_INSERT HT_5efabf307cb0, HASHTABLE_PROBE HT_5efabf332630, HASHTABLE_PROBE HT_5efabf350440, HASHTABLE_PROBE HT_5efabf354720, HASHTABLE_PROBE HT_5efabf3547e0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5efabf332630 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5efabf332630 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5efabf332630.for_each(KEY_5efabf332630, [&] __device__ (auto const SLOT_5efabf332630) {

auto const [slot_first5efabf332630, slot_second5efabf332630] = SLOT_5efabf332630;
if (!(true)) return;
uint64_t KEY_5efabf350440 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5efabf350440 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_5efabf350440.for_each(KEY_5efabf350440, [&] __device__ (auto const SLOT_5efabf350440) {

auto const [slot_first5efabf350440, slot_second5efabf350440] = SLOT_5efabf350440;
if (!(true)) return;
uint64_t KEY_5efabf354720 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5efabf354720 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5efabf354720.for_each(KEY_5efabf354720, [&] __device__ (auto const SLOT_5efabf354720) {

auto const [slot_first5efabf354720, slot_second5efabf354720] = SLOT_5efabf354720;
if (!(true)) return;
uint64_t KEY_5efabf3547e0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5efabf3547e0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5efabf3547e0.for_each(KEY_5efabf3547e0, [&] __device__ (auto const SLOT_5efabf3547e0) {

auto const [slot_first5efabf3547e0, slot_second5efabf3547e0] = SLOT_5efabf3547e0;
if (!(true)) return;
uint64_t KEY_5efabf307cb0 = 0;
auto reg_date__d_year = date__d_year[BUF_5efabf3547e0[slot_second5efabf3547e0 * 1 + 0]];

KEY_5efabf307cb0 |= reg_date__d_year;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[BUF_5efabf350440[slot_second5efabf350440 * 1 + 0]];
KEY_5efabf307cb0 <<= 16;
KEY_5efabf307cb0 |= reg_customer__c_nation_encoded;
//Create aggregation hash table
HT_5efabf307cb0.insert(cuco::pair{KEY_5efabf307cb0, 1});
});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5efabf3657e0(uint64_t* BUF_5efabf332630, uint64_t* BUF_5efabf350440, uint64_t* BUF_5efabf354720, uint64_t* BUF_5efabf3547e0, HASHTABLE_FIND HT_5efabf307cb0, HASHTABLE_PROBE HT_5efabf332630, HASHTABLE_PROBE HT_5efabf350440, HASHTABLE_PROBE HT_5efabf354720, HASHTABLE_PROBE HT_5efabf3547e0, DBI16Type* KEY_5efabf307cb0customer__c_nation_encoded, DBI32Type* KEY_5efabf307cb0date__d_year, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, DBDecimalType* lineorder__lo_supplycost, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5efabf332630 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5efabf332630 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5efabf332630.for_each(KEY_5efabf332630, [&] __device__ (auto const SLOT_5efabf332630) {
auto const [slot_first5efabf332630, slot_second5efabf332630] = SLOT_5efabf332630;
if (!(true)) return;
uint64_t KEY_5efabf350440 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_5efabf350440 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_5efabf350440.for_each(KEY_5efabf350440, [&] __device__ (auto const SLOT_5efabf350440) {
auto const [slot_first5efabf350440, slot_second5efabf350440] = SLOT_5efabf350440;
if (!(true)) return;
uint64_t KEY_5efabf354720 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5efabf354720 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5efabf354720.for_each(KEY_5efabf354720, [&] __device__ (auto const SLOT_5efabf354720) {
auto const [slot_first5efabf354720, slot_second5efabf354720] = SLOT_5efabf354720;
if (!(true)) return;
uint64_t KEY_5efabf3547e0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5efabf3547e0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5efabf3547e0.for_each(KEY_5efabf3547e0, [&] __device__ (auto const SLOT_5efabf3547e0) {
auto const [slot_first5efabf3547e0, slot_second5efabf3547e0] = SLOT_5efabf3547e0;
if (!(true)) return;
uint64_t KEY_5efabf307cb0 = 0;
auto reg_date__d_year = date__d_year[BUF_5efabf3547e0[slot_second5efabf3547e0 * 1 + 0]];

KEY_5efabf307cb0 |= reg_date__d_year;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[BUF_5efabf350440[slot_second5efabf350440 * 1 + 0]];
KEY_5efabf307cb0 <<= 16;
KEY_5efabf307cb0 |= reg_customer__c_nation_encoded;
//Aggregate in hashtable
auto buf_idx_5efabf307cb0 = HT_5efabf307cb0.find(KEY_5efabf307cb0)->second;
auto reg_lineorder__lo_supplycost = lineorder__lo_supplycost[tid];
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
auto reg_map0__tmp_attr1 = (reg_lineorder__lo_revenue) - (reg_lineorder__lo_supplycost);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5efabf307cb0], reg_map0__tmp_attr1);
KEY_5efabf307cb0date__d_year[buf_idx_5efabf307cb0] = reg_date__d_year;
KEY_5efabf307cb0customer__c_nation_encoded[buf_idx_5efabf307cb0] = reg_customer__c_nation_encoded;
});
});
});
});
}
__global__ void count_5efabf378b50(uint64_t* COUNT5efabf2e7da0, size_t COUNT5efabf307cb0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5efabf307cb0) return;
//Materialize count
atomicAdd((int*)COUNT5efabf2e7da0, 1);
}
__global__ void main_5efabf378b50(size_t COUNT5efabf307cb0, DBDecimalType* MAT5efabf2e7da0aggr0__tmp_attr0, DBI16Type* MAT5efabf2e7da0customer__c_nation_encoded, DBI32Type* MAT5efabf2e7da0date__d_year, uint64_t* MAT_IDX5efabf2e7da0, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5efabf307cb0) return;
//Materialize buffers
auto mat_idx5efabf2e7da0 = atomicAdd((int*)MAT_IDX5efabf2e7da0, 1);
auto reg_date__d_year = date__d_year[tid];
MAT5efabf2e7da0date__d_year[mat_idx5efabf2e7da0] = reg_date__d_year;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[tid];
MAT5efabf2e7da0customer__c_nation_encoded[mat_idx5efabf2e7da0] = reg_customer__c_nation_encoded;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5efabf2e7da0aggr0__tmp_attr0[mat_idx5efabf2e7da0] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT5efabf332630;
hipMalloc(&d_COUNT5efabf332630, sizeof(uint64_t));
hipMemset(d_COUNT5efabf332630, 0, sizeof(uint64_t));
count_5efabf2f9580<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT5efabf332630, d_supplier__s_region, supplier_size);
uint64_t COUNT5efabf332630;
hipMemcpy(&COUNT5efabf332630, d_COUNT5efabf332630, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5efabf332630;
hipMalloc(&d_BUF_IDX_5efabf332630, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5efabf332630, 0, sizeof(uint64_t));
uint64_t* d_BUF_5efabf332630;
hipMalloc(&d_BUF_5efabf332630, sizeof(uint64_t) * COUNT5efabf332630 * 1);
auto d_HT_5efabf332630 = cuco::experimental::static_multimap{ (int)COUNT5efabf332630*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5efabf2f9580<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_5efabf332630, d_BUF_IDX_5efabf332630, d_HT_5efabf332630.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5efabf350440;
hipMalloc(&d_COUNT5efabf350440, sizeof(uint64_t));
hipMemset(d_COUNT5efabf350440, 0, sizeof(uint64_t));
count_5efabf2f97c0<<<std::ceil((float)customer_size/128.), 128>>>(d_COUNT5efabf350440, d_customer__c_region, customer_size);
uint64_t COUNT5efabf350440;
hipMemcpy(&COUNT5efabf350440, d_COUNT5efabf350440, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5efabf350440;
hipMalloc(&d_BUF_IDX_5efabf350440, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5efabf350440, 0, sizeof(uint64_t));
uint64_t* d_BUF_5efabf350440;
hipMalloc(&d_BUF_5efabf350440, sizeof(uint64_t) * COUNT5efabf350440 * 1);
auto d_HT_5efabf350440 = cuco::experimental::static_multimap{ (int)COUNT5efabf350440*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5efabf2f97c0<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_5efabf350440, d_BUF_IDX_5efabf350440, d_HT_5efabf350440.ref(cuco::insert), d_customer__c_custkey, d_customer__c_region, customer_size);
//Materialize count
uint64_t* d_COUNT5efabf354720;
hipMalloc(&d_COUNT5efabf354720, sizeof(uint64_t));
hipMemset(d_COUNT5efabf354720, 0, sizeof(uint64_t));
count_5efabf362f80<<<std::ceil((float)part_size/128.), 128>>>(d_COUNT5efabf354720, d_part__p_mfgr, part_size);
uint64_t COUNT5efabf354720;
hipMemcpy(&COUNT5efabf354720, d_COUNT5efabf354720, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5efabf354720;
hipMalloc(&d_BUF_IDX_5efabf354720, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5efabf354720, 0, sizeof(uint64_t));
uint64_t* d_BUF_5efabf354720;
hipMalloc(&d_BUF_5efabf354720, sizeof(uint64_t) * COUNT5efabf354720 * 1);
auto d_HT_5efabf354720 = cuco::experimental::static_multimap{ (int)COUNT5efabf354720*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5efabf362f80<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_5efabf354720, d_BUF_IDX_5efabf354720, d_HT_5efabf354720.ref(cuco::insert), d_part__p_mfgr, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT5efabf3547e0;
hipMalloc(&d_COUNT5efabf3547e0, sizeof(uint64_t));
hipMemset(d_COUNT5efabf3547e0, 0, sizeof(uint64_t));
count_5efabf352b40<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT5efabf3547e0, date_size);
uint64_t COUNT5efabf3547e0;
hipMemcpy(&COUNT5efabf3547e0, d_COUNT5efabf3547e0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5efabf3547e0;
hipMalloc(&d_BUF_IDX_5efabf3547e0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5efabf3547e0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5efabf3547e0;
hipMalloc(&d_BUF_5efabf3547e0, sizeof(uint64_t) * COUNT5efabf3547e0 * 1);
auto d_HT_5efabf3547e0 = cuco::experimental::static_multimap{ (int)COUNT5efabf3547e0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5efabf352b40<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_5efabf3547e0, d_BUF_IDX_5efabf3547e0, d_HT_5efabf3547e0.ref(cuco::insert), d_date__d_datekey, date_size);
//Create aggregation hash table
auto d_HT_5efabf307cb0 = cuco::static_map{ (int)87950*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5efabf3657e0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5efabf332630, d_BUF_5efabf350440, d_BUF_5efabf354720, d_BUF_5efabf3547e0, d_HT_5efabf307cb0.ref(cuco::insert), d_HT_5efabf332630.ref(cuco::for_each), d_HT_5efabf350440.ref(cuco::for_each), d_HT_5efabf354720.ref(cuco::for_each), d_HT_5efabf3547e0.ref(cuco::for_each), d_customer__c_nation_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
size_t COUNT5efabf307cb0 = d_HT_5efabf307cb0.size();
thrust::device_vector<int64_t> keys_5efabf307cb0(COUNT5efabf307cb0), vals_5efabf307cb0(COUNT5efabf307cb0);
d_HT_5efabf307cb0.retrieve_all(keys_5efabf307cb0.begin(), vals_5efabf307cb0.begin());
d_HT_5efabf307cb0.clear();
int64_t* raw_keys5efabf307cb0 = thrust::raw_pointer_cast(keys_5efabf307cb0.data());
insertKeys<<<std::ceil((float)COUNT5efabf307cb0/128.), 128>>>(raw_keys5efabf307cb0, d_HT_5efabf307cb0.ref(cuco::insert), COUNT5efabf307cb0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5efabf307cb0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5efabf307cb0);
DBI32Type* d_KEY_5efabf307cb0date__d_year;
hipMalloc(&d_KEY_5efabf307cb0date__d_year, sizeof(DBI32Type) * COUNT5efabf307cb0);
hipMemset(d_KEY_5efabf307cb0date__d_year, 0, sizeof(DBI32Type) * COUNT5efabf307cb0);
DBI16Type* d_KEY_5efabf307cb0customer__c_nation_encoded;
hipMalloc(&d_KEY_5efabf307cb0customer__c_nation_encoded, sizeof(DBI16Type) * COUNT5efabf307cb0);
hipMemset(d_KEY_5efabf307cb0customer__c_nation_encoded, 0, sizeof(DBI16Type) * COUNT5efabf307cb0);
main_5efabf3657e0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5efabf332630, d_BUF_5efabf350440, d_BUF_5efabf354720, d_BUF_5efabf3547e0, d_HT_5efabf307cb0.ref(cuco::find), d_HT_5efabf332630.ref(cuco::for_each), d_HT_5efabf350440.ref(cuco::for_each), d_HT_5efabf354720.ref(cuco::for_each), d_HT_5efabf3547e0.ref(cuco::for_each), d_KEY_5efabf307cb0customer__c_nation_encoded, d_KEY_5efabf307cb0date__d_year, d_aggr0__tmp_attr0, d_customer__c_nation_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, d_lineorder__lo_supplycost, lineorder_size);
//Materialize count
uint64_t* d_COUNT5efabf2e7da0;
hipMalloc(&d_COUNT5efabf2e7da0, sizeof(uint64_t));
hipMemset(d_COUNT5efabf2e7da0, 0, sizeof(uint64_t));
count_5efabf378b50<<<std::ceil((float)COUNT5efabf307cb0/128.), 128>>>(d_COUNT5efabf2e7da0, COUNT5efabf307cb0);
uint64_t COUNT5efabf2e7da0;
hipMemcpy(&COUNT5efabf2e7da0, d_COUNT5efabf2e7da0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5efabf2e7da0;
hipMalloc(&d_MAT_IDX5efabf2e7da0, sizeof(uint64_t));
hipMemset(d_MAT_IDX5efabf2e7da0, 0, sizeof(uint64_t));
auto MAT5efabf2e7da0date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5efabf2e7da0);
DBI32Type* d_MAT5efabf2e7da0date__d_year;
hipMalloc(&d_MAT5efabf2e7da0date__d_year, sizeof(DBI32Type) * COUNT5efabf2e7da0);
auto MAT5efabf2e7da0customer__c_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5efabf2e7da0);
DBI16Type* d_MAT5efabf2e7da0customer__c_nation_encoded;
hipMalloc(&d_MAT5efabf2e7da0customer__c_nation_encoded, sizeof(DBI16Type) * COUNT5efabf2e7da0);
auto MAT5efabf2e7da0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5efabf2e7da0);
DBDecimalType* d_MAT5efabf2e7da0aggr0__tmp_attr0;
hipMalloc(&d_MAT5efabf2e7da0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5efabf2e7da0);
main_5efabf378b50<<<std::ceil((float)COUNT5efabf307cb0/128.), 128>>>(COUNT5efabf307cb0, d_MAT5efabf2e7da0aggr0__tmp_attr0, d_MAT5efabf2e7da0customer__c_nation_encoded, d_MAT5efabf2e7da0date__d_year, d_MAT_IDX5efabf2e7da0, d_aggr0__tmp_attr0, d_KEY_5efabf307cb0customer__c_nation_encoded, d_KEY_5efabf307cb0date__d_year);
hipMemcpy(MAT5efabf2e7da0date__d_year, d_MAT5efabf2e7da0date__d_year, sizeof(DBI32Type) * COUNT5efabf2e7da0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5efabf2e7da0customer__c_nation_encoded, d_MAT5efabf2e7da0customer__c_nation_encoded, sizeof(DBI16Type) * COUNT5efabf2e7da0, hipMemcpyDeviceToHost);
hipMemcpy(MAT5efabf2e7da0aggr0__tmp_attr0, d_MAT5efabf2e7da0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5efabf2e7da0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5efabf2e7da0; i++) { std::cout << MAT5efabf2e7da0date__d_year[i] << "\t";
std::cout << customer__c_nation_map[MAT5efabf2e7da0customer__c_nation_encoded[i]] << "\t";
std::cout << MAT5efabf2e7da0aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_5efabf332630);
hipFree(d_BUF_IDX_5efabf332630);
hipFree(d_COUNT5efabf332630);
hipFree(d_BUF_5efabf350440);
hipFree(d_BUF_IDX_5efabf350440);
hipFree(d_COUNT5efabf350440);
hipFree(d_BUF_5efabf354720);
hipFree(d_BUF_IDX_5efabf354720);
hipFree(d_COUNT5efabf354720);
hipFree(d_BUF_5efabf3547e0);
hipFree(d_BUF_IDX_5efabf3547e0);
hipFree(d_COUNT5efabf3547e0);
hipFree(d_KEY_5efabf307cb0customer__c_nation_encoded);
hipFree(d_KEY_5efabf307cb0date__d_year);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5efabf2e7da0);
hipFree(d_MAT5efabf2e7da0aggr0__tmp_attr0);
hipFree(d_MAT5efabf2e7da0customer__c_nation_encoded);
hipFree(d_MAT5efabf2e7da0date__d_year);
hipFree(d_MAT_IDX5efabf2e7da0);
free(MAT5efabf2e7da0aggr0__tmp_attr0);
free(MAT5efabf2e7da0customer__c_nation_encoded);
free(MAT5efabf2e7da0date__d_year);
}