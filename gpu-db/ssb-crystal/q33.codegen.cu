#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_57e2860267c0(uint64_t* COUNT57e28603d030, DBStringType* customer__c_city, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_city = customer__c_city[tid];
if (!((evaluatePredicate(reg_customer__c_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_customer__c_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT57e28603d030, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_57e2860267c0(uint64_t* BUF_57e28603d030, uint64_t* BUF_IDX_57e28603d030, HASHTABLE_INSERT HT_57e28603d030, DBStringType* customer__c_city, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_city = customer__c_city[tid];
if (!((evaluatePredicate(reg_customer__c_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_customer__c_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_57e28603d030 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_57e28603d030 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_57e28603d030 = atomicAdd((int*)BUF_IDX_57e28603d030, 1);
HT_57e28603d030.insert(cuco::pair{KEY_57e28603d030, buf_idx_57e28603d030});
BUF_57e28603d030[buf_idx_57e28603d030 * 1 + 0] = tid;
}
__global__ void count_57e286026d60(uint64_t* COUNT57e286044b50, DBStringType* supplier__s_city, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_city = supplier__s_city[tid];
if (!((evaluatePredicate(reg_supplier__s_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_supplier__s_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT57e286044b50, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_57e286026d60(uint64_t* BUF_57e286044b50, uint64_t* BUF_IDX_57e286044b50, HASHTABLE_INSERT HT_57e286044b50, DBStringType* supplier__s_city, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_city = supplier__s_city[tid];
if (!((evaluatePredicate(reg_supplier__s_city, "UNITED KI1", Predicate::eq)) || (evaluatePredicate(reg_supplier__s_city, "UNITED KI5", Predicate::eq)))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_57e286044b50 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_57e286044b50 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_57e286044b50 = atomicAdd((int*)BUF_IDX_57e286044b50, 1);
HT_57e286044b50.insert(cuco::pair{KEY_57e286044b50, buf_idx_57e286044b50});
BUF_57e286044b50[buf_idx_57e286044b50 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_57e2860408c0(uint64_t* BUF_57e28603d030, uint64_t* BUF_57e286044b50, uint64_t* COUNT57e28603cef0, HASHTABLE_PROBE HT_57e28603d030, HASHTABLE_PROBE HT_57e286044b50, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_57e28603d030 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_57e28603d030 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_57e28603d030.for_each(KEY_57e28603d030, [&] __device__ (auto const SLOT_57e28603d030) {

auto const [slot_first57e28603d030, slot_second57e28603d030] = SLOT_57e28603d030;
if (!(true)) return;
uint64_t KEY_57e286044b50 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_57e286044b50 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_57e286044b50.for_each(KEY_57e286044b50, [&] __device__ (auto const SLOT_57e286044b50) {

auto const [slot_first57e286044b50, slot_second57e286044b50] = SLOT_57e286044b50;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT57e28603cef0, 1);
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_57e2860408c0(uint64_t* BUF_57e28603cef0, uint64_t* BUF_57e28603d030, uint64_t* BUF_57e286044b50, uint64_t* BUF_IDX_57e28603cef0, HASHTABLE_INSERT HT_57e28603cef0, HASHTABLE_PROBE HT_57e28603d030, HASHTABLE_PROBE HT_57e286044b50, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_57e28603d030 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_57e28603d030 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_57e28603d030.for_each(KEY_57e28603d030, [&] __device__ (auto const SLOT_57e28603d030) {
auto const [slot_first57e28603d030, slot_second57e28603d030] = SLOT_57e28603d030;
if (!(true)) return;
uint64_t KEY_57e286044b50 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_57e286044b50 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_57e286044b50.for_each(KEY_57e286044b50, [&] __device__ (auto const SLOT_57e286044b50) {
auto const [slot_first57e286044b50, slot_second57e286044b50] = SLOT_57e286044b50;
if (!(true)) return;
uint64_t KEY_57e28603cef0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_57e28603cef0 |= reg_lineorder__lo_orderdate;
// Insert hash table kernel;
auto buf_idx_57e28603cef0 = atomicAdd((int*)BUF_IDX_57e28603cef0, 1);
HT_57e28603cef0.insert(cuco::pair{KEY_57e28603cef0, buf_idx_57e28603cef0});
BUF_57e28603cef0[buf_idx_57e28603cef0 * 3 + 0] = BUF_57e28603d030[slot_second57e28603d030 * 1 + 0];
BUF_57e28603cef0[buf_idx_57e28603cef0 * 3 + 1] = tid;
BUF_57e28603cef0[buf_idx_57e28603cef0 * 3 + 2] = BUF_57e286044b50[slot_second57e286044b50 * 1 + 0];
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_57e286051ac0(uint64_t* BUF_57e28603cef0, HASHTABLE_INSERT HT_57e285ff5ec0, HASHTABLE_PROBE HT_57e28603cef0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_57e28603cef0 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_57e28603cef0 |= reg_date__d_datekey;
//Probe Hash table
HT_57e28603cef0.for_each(KEY_57e28603cef0, [&] __device__ (auto const SLOT_57e28603cef0) {

auto const [slot_first57e28603cef0, slot_second57e28603cef0] = SLOT_57e28603cef0;
if (!(true)) return;
uint64_t KEY_57e285ff5ec0 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_57e28603cef0[slot_second57e28603cef0 * 3 + 0]];

KEY_57e285ff5ec0 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_57e28603cef0[slot_second57e28603cef0 * 3 + 2]];
KEY_57e285ff5ec0 <<= 16;
KEY_57e285ff5ec0 |= reg_supplier__s_city_encoded;
KEY_57e285ff5ec0 <<= 32;
KEY_57e285ff5ec0 |= reg_date__d_year;
//Create aggregation hash table
HT_57e285ff5ec0.insert(cuco::pair{KEY_57e285ff5ec0, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_57e286051ac0(uint64_t* BUF_57e28603cef0, HASHTABLE_FIND HT_57e285ff5ec0, HASHTABLE_PROBE HT_57e28603cef0, DBI16Type* KEY_57e285ff5ec0customer__c_city_encoded, DBI32Type* KEY_57e285ff5ec0date__d_year, DBI16Type* KEY_57e285ff5ec0supplier__s_city_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size, DBDecimalType* lineorder__lo_revenue, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_57e28603cef0 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_57e28603cef0 |= reg_date__d_datekey;
//Probe Hash table
HT_57e28603cef0.for_each(KEY_57e28603cef0, [&] __device__ (auto const SLOT_57e28603cef0) {
auto const [slot_first57e28603cef0, slot_second57e28603cef0] = SLOT_57e28603cef0;
if (!(true)) return;
uint64_t KEY_57e285ff5ec0 = 0;
auto reg_customer__c_city_encoded = customer__c_city_encoded[BUF_57e28603cef0[slot_second57e28603cef0 * 3 + 0]];

KEY_57e285ff5ec0 |= reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[BUF_57e28603cef0[slot_second57e28603cef0 * 3 + 2]];
KEY_57e285ff5ec0 <<= 16;
KEY_57e285ff5ec0 |= reg_supplier__s_city_encoded;
KEY_57e285ff5ec0 <<= 32;
KEY_57e285ff5ec0 |= reg_date__d_year;
//Aggregate in hashtable
auto buf_idx_57e285ff5ec0 = HT_57e285ff5ec0.find(KEY_57e285ff5ec0)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_57e28603cef0[slot_second57e28603cef0 * 3 + 1]];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_57e285ff5ec0], reg_lineorder__lo_revenue);
KEY_57e285ff5ec0customer__c_city_encoded[buf_idx_57e285ff5ec0] = reg_customer__c_city_encoded;
KEY_57e285ff5ec0supplier__s_city_encoded[buf_idx_57e285ff5ec0] = reg_supplier__s_city_encoded;
KEY_57e285ff5ec0date__d_year[buf_idx_57e285ff5ec0] = reg_date__d_year;
});
}
__global__ void count_57e286060d80(size_t COUNT57e285ff5ec0, uint64_t* COUNT57e2860093c0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT57e285ff5ec0) return;
//Materialize count
atomicAdd((int*)COUNT57e2860093c0, 1);
}
__global__ void main_57e286060d80(size_t COUNT57e285ff5ec0, DBDecimalType* MAT57e2860093c0aggr0__tmp_attr0, DBI16Type* MAT57e2860093c0customer__c_city_encoded, DBI32Type* MAT57e2860093c0date__d_year, DBI16Type* MAT57e2860093c0supplier__s_city_encoded, uint64_t* MAT_IDX57e2860093c0, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_city_encoded, DBI32Type* date__d_year, DBI16Type* supplier__s_city_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT57e285ff5ec0) return;
//Materialize buffers
auto mat_idx57e2860093c0 = atomicAdd((int*)MAT_IDX57e2860093c0, 1);
auto reg_customer__c_city_encoded = customer__c_city_encoded[tid];
MAT57e2860093c0customer__c_city_encoded[mat_idx57e2860093c0] = reg_customer__c_city_encoded;
auto reg_supplier__s_city_encoded = supplier__s_city_encoded[tid];
MAT57e2860093c0supplier__s_city_encoded[mat_idx57e2860093c0] = reg_supplier__s_city_encoded;
auto reg_date__d_year = date__d_year[tid];
MAT57e2860093c0date__d_year[mat_idx57e2860093c0] = reg_date__d_year;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT57e2860093c0aggr0__tmp_attr0[mat_idx57e2860093c0] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT57e28603d030;
hipMalloc(&d_COUNT57e28603d030, sizeof(uint64_t));
hipMemset(d_COUNT57e28603d030, 0, sizeof(uint64_t));
count_57e2860267c0<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT57e28603d030, d_customer__c_city, customer_size);
uint64_t COUNT57e28603d030;
hipMemcpy(&COUNT57e28603d030, d_COUNT57e28603d030, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_57e28603d030;
hipMalloc(&d_BUF_IDX_57e28603d030, sizeof(uint64_t));
hipMemset(d_BUF_IDX_57e28603d030, 0, sizeof(uint64_t));
uint64_t* d_BUF_57e28603d030;
hipMalloc(&d_BUF_57e28603d030, sizeof(uint64_t) * COUNT57e28603d030 * 1);
auto d_HT_57e28603d030 = cuco::experimental::static_multimap{ (int)COUNT57e28603d030*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_57e2860267c0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_57e28603d030, d_BUF_IDX_57e28603d030, d_HT_57e28603d030.ref(cuco::insert), d_customer__c_city, d_customer__c_custkey, customer_size);
//Materialize count
uint64_t* d_COUNT57e286044b50;
hipMalloc(&d_COUNT57e286044b50, sizeof(uint64_t));
hipMemset(d_COUNT57e286044b50, 0, sizeof(uint64_t));
count_57e286026d60<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT57e286044b50, d_supplier__s_city, supplier_size);
uint64_t COUNT57e286044b50;
hipMemcpy(&COUNT57e286044b50, d_COUNT57e286044b50, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_57e286044b50;
hipMalloc(&d_BUF_IDX_57e286044b50, sizeof(uint64_t));
hipMemset(d_BUF_IDX_57e286044b50, 0, sizeof(uint64_t));
uint64_t* d_BUF_57e286044b50;
hipMalloc(&d_BUF_57e286044b50, sizeof(uint64_t) * COUNT57e286044b50 * 1);
auto d_HT_57e286044b50 = cuco::experimental::static_multimap{ (int)COUNT57e286044b50*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_57e286026d60<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_57e286044b50, d_BUF_IDX_57e286044b50, d_HT_57e286044b50.ref(cuco::insert), d_supplier__s_city, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT57e28603cef0;
hipMalloc(&d_COUNT57e28603cef0, sizeof(uint64_t));
hipMemset(d_COUNT57e28603cef0, 0, sizeof(uint64_t));
count_57e2860408c0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_57e28603d030, d_BUF_57e286044b50, d_COUNT57e28603cef0, d_HT_57e28603d030.ref(cuco::for_each), d_HT_57e286044b50.ref(cuco::for_each), d_lineorder__lo_custkey, d_lineorder__lo_suppkey, lineorder_size);
uint64_t COUNT57e28603cef0;
hipMemcpy(&COUNT57e28603cef0, d_COUNT57e28603cef0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_57e28603cef0;
hipMalloc(&d_BUF_IDX_57e28603cef0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_57e28603cef0, 0, sizeof(uint64_t));
uint64_t* d_BUF_57e28603cef0;
hipMalloc(&d_BUF_57e28603cef0, sizeof(uint64_t) * COUNT57e28603cef0 * 3);
auto d_HT_57e28603cef0 = cuco::experimental::static_multimap{ (int)COUNT57e28603cef0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_57e2860408c0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_57e28603cef0, d_BUF_57e28603d030, d_BUF_57e286044b50, d_BUF_IDX_57e28603cef0, d_HT_57e28603cef0.ref(cuco::insert), d_HT_57e28603d030.ref(cuco::for_each), d_HT_57e286044b50.ref(cuco::for_each), d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size);
//Create aggregation hash table
auto d_HT_57e285ff5ec0 = cuco::static_map{ (int)132*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_57e286051ac0<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_57e28603cef0, d_HT_57e285ff5ec0.ref(cuco::insert), d_HT_57e28603cef0.ref(cuco::for_each), d_customer__c_city_encoded, d_date__d_datekey, d_date__d_year, date_size, d_supplier__s_city_encoded);
size_t COUNT57e285ff5ec0 = d_HT_57e285ff5ec0.size();
thrust::device_vector<int64_t> keys_57e285ff5ec0(COUNT57e285ff5ec0), vals_57e285ff5ec0(COUNT57e285ff5ec0);
d_HT_57e285ff5ec0.retrieve_all(keys_57e285ff5ec0.begin(), vals_57e285ff5ec0.begin());
d_HT_57e285ff5ec0.clear();
int64_t* raw_keys57e285ff5ec0 = thrust::raw_pointer_cast(keys_57e285ff5ec0.data());
insertKeys<<<std::ceil((float)COUNT57e285ff5ec0/32.), 32>>>(raw_keys57e285ff5ec0, d_HT_57e285ff5ec0.ref(cuco::insert), COUNT57e285ff5ec0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT57e285ff5ec0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT57e285ff5ec0);
DBI16Type* d_KEY_57e285ff5ec0customer__c_city_encoded;
hipMalloc(&d_KEY_57e285ff5ec0customer__c_city_encoded, sizeof(DBI16Type) * COUNT57e285ff5ec0);
hipMemset(d_KEY_57e285ff5ec0customer__c_city_encoded, 0, sizeof(DBI16Type) * COUNT57e285ff5ec0);
DBI16Type* d_KEY_57e285ff5ec0supplier__s_city_encoded;
hipMalloc(&d_KEY_57e285ff5ec0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT57e285ff5ec0);
hipMemset(d_KEY_57e285ff5ec0supplier__s_city_encoded, 0, sizeof(DBI16Type) * COUNT57e285ff5ec0);
DBI32Type* d_KEY_57e285ff5ec0date__d_year;
hipMalloc(&d_KEY_57e285ff5ec0date__d_year, sizeof(DBI32Type) * COUNT57e285ff5ec0);
hipMemset(d_KEY_57e285ff5ec0date__d_year, 0, sizeof(DBI32Type) * COUNT57e285ff5ec0);
main_57e286051ac0<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_57e28603cef0, d_HT_57e285ff5ec0.ref(cuco::find), d_HT_57e28603cef0.ref(cuco::for_each), d_KEY_57e285ff5ec0customer__c_city_encoded, d_KEY_57e285ff5ec0date__d_year, d_KEY_57e285ff5ec0supplier__s_city_encoded, d_aggr0__tmp_attr0, d_customer__c_city_encoded, d_date__d_datekey, d_date__d_year, date_size, d_lineorder__lo_revenue, d_supplier__s_city_encoded);
//Materialize count
uint64_t* d_COUNT57e2860093c0;
hipMalloc(&d_COUNT57e2860093c0, sizeof(uint64_t));
hipMemset(d_COUNT57e2860093c0, 0, sizeof(uint64_t));
count_57e286060d80<<<std::ceil((float)COUNT57e285ff5ec0/32.), 32>>>(COUNT57e285ff5ec0, d_COUNT57e2860093c0);
uint64_t COUNT57e2860093c0;
hipMemcpy(&COUNT57e2860093c0, d_COUNT57e2860093c0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX57e2860093c0;
hipMalloc(&d_MAT_IDX57e2860093c0, sizeof(uint64_t));
hipMemset(d_MAT_IDX57e2860093c0, 0, sizeof(uint64_t));
auto MAT57e2860093c0customer__c_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT57e2860093c0);
DBI16Type* d_MAT57e2860093c0customer__c_city_encoded;
hipMalloc(&d_MAT57e2860093c0customer__c_city_encoded, sizeof(DBI16Type) * COUNT57e2860093c0);
auto MAT57e2860093c0supplier__s_city_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT57e2860093c0);
DBI16Type* d_MAT57e2860093c0supplier__s_city_encoded;
hipMalloc(&d_MAT57e2860093c0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT57e2860093c0);
auto MAT57e2860093c0date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT57e2860093c0);
DBI32Type* d_MAT57e2860093c0date__d_year;
hipMalloc(&d_MAT57e2860093c0date__d_year, sizeof(DBI32Type) * COUNT57e2860093c0);
auto MAT57e2860093c0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT57e2860093c0);
DBDecimalType* d_MAT57e2860093c0aggr0__tmp_attr0;
hipMalloc(&d_MAT57e2860093c0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT57e2860093c0);
main_57e286060d80<<<std::ceil((float)COUNT57e285ff5ec0/32.), 32>>>(COUNT57e285ff5ec0, d_MAT57e2860093c0aggr0__tmp_attr0, d_MAT57e2860093c0customer__c_city_encoded, d_MAT57e2860093c0date__d_year, d_MAT57e2860093c0supplier__s_city_encoded, d_MAT_IDX57e2860093c0, d_aggr0__tmp_attr0, d_KEY_57e285ff5ec0customer__c_city_encoded, d_KEY_57e285ff5ec0date__d_year, d_KEY_57e285ff5ec0supplier__s_city_encoded);
hipMemcpy(MAT57e2860093c0customer__c_city_encoded, d_MAT57e2860093c0customer__c_city_encoded, sizeof(DBI16Type) * COUNT57e2860093c0, hipMemcpyDeviceToHost);
hipMemcpy(MAT57e2860093c0supplier__s_city_encoded, d_MAT57e2860093c0supplier__s_city_encoded, sizeof(DBI16Type) * COUNT57e2860093c0, hipMemcpyDeviceToHost);
hipMemcpy(MAT57e2860093c0date__d_year, d_MAT57e2860093c0date__d_year, sizeof(DBI32Type) * COUNT57e2860093c0, hipMemcpyDeviceToHost);
hipMemcpy(MAT57e2860093c0aggr0__tmp_attr0, d_MAT57e2860093c0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT57e2860093c0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT57e2860093c0; i++) { std::cout << customer__c_city_map[MAT57e2860093c0customer__c_city_encoded[i]] << "\t";
std::cout << supplier__s_city_map[MAT57e2860093c0supplier__s_city_encoded[i]] << "\t";
std::cout << MAT57e2860093c0date__d_year[i] << "\t";
std::cout << MAT57e2860093c0aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_57e28603d030);
hipFree(d_BUF_IDX_57e28603d030);
hipFree(d_COUNT57e28603d030);
hipFree(d_BUF_57e286044b50);
hipFree(d_BUF_IDX_57e286044b50);
hipFree(d_COUNT57e286044b50);
hipFree(d_BUF_57e28603cef0);
hipFree(d_BUF_IDX_57e28603cef0);
hipFree(d_COUNT57e28603cef0);
hipFree(d_KEY_57e285ff5ec0customer__c_city_encoded);
hipFree(d_KEY_57e285ff5ec0date__d_year);
hipFree(d_KEY_57e285ff5ec0supplier__s_city_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT57e2860093c0);
hipFree(d_MAT57e2860093c0aggr0__tmp_attr0);
hipFree(d_MAT57e2860093c0customer__c_city_encoded);
hipFree(d_MAT57e2860093c0date__d_year);
hipFree(d_MAT57e2860093c0supplier__s_city_encoded);
hipFree(d_MAT_IDX57e2860093c0);
free(MAT57e2860093c0aggr0__tmp_attr0);
free(MAT57e2860093c0customer__c_city_encoded);
free(MAT57e2860093c0date__d_year);
free(MAT57e2860093c0supplier__s_city_encoded);
}