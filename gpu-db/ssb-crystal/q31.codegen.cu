#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_601cc56dc0f0(uint64_t* COUNT601cc56f72a0, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT601cc56f72a0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_601cc56dc0f0(uint64_t* BUF_601cc56f72a0, uint64_t* BUF_IDX_601cc56f72a0, HASHTABLE_INSERT HT_601cc56f72a0, DBI32Type* customer__c_custkey, DBStringType* customer__c_region, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_region = customer__c_region[tid];
if (!(evaluatePredicate(reg_customer__c_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_601cc56f72a0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_601cc56f72a0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_601cc56f72a0 = atomicAdd((int*)BUF_IDX_601cc56f72a0, 1);
HT_601cc56f72a0.insert(cuco::pair{KEY_601cc56f72a0, buf_idx_601cc56f72a0});
BUF_601cc56f72a0[buf_idx_601cc56f72a0 * 1 + 0] = tid;
}
__global__ void count_601cc5702b80(uint64_t* COUNT601cc56f9a90, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT601cc56f9a90, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_601cc5702b80(uint64_t* BUF_601cc56f9a90, uint64_t* BUF_IDX_601cc56f9a90, HASHTABLE_INSERT HT_601cc56f9a90, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "ASIA", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_601cc56f9a90 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_601cc56f9a90 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_601cc56f9a90 = atomicAdd((int*)BUF_IDX_601cc56f9a90, 1);
HT_601cc56f9a90.insert(cuco::pair{KEY_601cc56f9a90, buf_idx_601cc56f9a90});
BUF_601cc56f9a90[buf_idx_601cc56f9a90 * 1 + 0] = tid;
}
__global__ void count_601cc57089f0(uint64_t* COUNT601cc56dd2e0, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT601cc56dd2e0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_601cc57089f0(uint64_t* BUF_601cc56dd2e0, uint64_t* BUF_IDX_601cc56dd2e0, HASHTABLE_INSERT HT_601cc56dd2e0, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1992, Predicate::gte) && evaluatePredicate(reg_date__d_year, 1997, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_601cc56dd2e0 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_601cc56dd2e0 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_601cc56dd2e0 = atomicAdd((int*)BUF_IDX_601cc56dd2e0, 1);
HT_601cc56dd2e0.insert(cuco::pair{KEY_601cc56dd2e0, buf_idx_601cc56dd2e0});
BUF_601cc56dd2e0[buf_idx_601cc56dd2e0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_601cc56dc690(uint64_t* BUF_601cc56dd2e0, uint64_t* BUF_601cc56f72a0, uint64_t* BUF_601cc56f9a90, HASHTABLE_INSERT HT_601cc56abd30, HASHTABLE_PROBE HT_601cc56dd2e0, HASHTABLE_PROBE HT_601cc56f72a0, HASHTABLE_PROBE HT_601cc56f9a90, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_601cc56f72a0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_601cc56f72a0 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_601cc56f72a0.for_each(KEY_601cc56f72a0, [&] __device__ (auto const SLOT_601cc56f72a0) {

auto const [slot_first601cc56f72a0, slot_second601cc56f72a0] = SLOT_601cc56f72a0;
if (!(true)) return;
uint64_t KEY_601cc56f9a90 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_601cc56f9a90 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_601cc56f9a90.for_each(KEY_601cc56f9a90, [&] __device__ (auto const SLOT_601cc56f9a90) {

auto const [slot_first601cc56f9a90, slot_second601cc56f9a90] = SLOT_601cc56f9a90;
if (!(true)) return;
uint64_t KEY_601cc56dd2e0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_601cc56dd2e0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_601cc56dd2e0.for_each(KEY_601cc56dd2e0, [&] __device__ (auto const SLOT_601cc56dd2e0) {

auto const [slot_first601cc56dd2e0, slot_second601cc56dd2e0] = SLOT_601cc56dd2e0;
if (!(true)) return;
uint64_t KEY_601cc56abd30 = 0;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[BUF_601cc56f72a0[slot_second601cc56f72a0 * 1 + 0]];

KEY_601cc56abd30 |= reg_customer__c_nation_encoded;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_601cc56f9a90[slot_second601cc56f9a90 * 1 + 0]];
KEY_601cc56abd30 <<= 16;
KEY_601cc56abd30 |= reg_supplier__s_nation_encoded;
auto reg_date__d_year = date__d_year[BUF_601cc56dd2e0[slot_second601cc56dd2e0 * 1 + 0]];
KEY_601cc56abd30 <<= 32;
KEY_601cc56abd30 |= reg_date__d_year;
//Create aggregation hash table
HT_601cc56abd30.insert(cuco::pair{KEY_601cc56abd30, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_601cc56dc690(uint64_t* BUF_601cc56dd2e0, uint64_t* BUF_601cc56f72a0, uint64_t* BUF_601cc56f9a90, HASHTABLE_FIND HT_601cc56abd30, HASHTABLE_PROBE HT_601cc56dd2e0, HASHTABLE_PROBE HT_601cc56f72a0, HASHTABLE_PROBE HT_601cc56f9a90, DBI16Type* KEY_601cc56abd30customer__c_nation_encoded, DBI32Type* KEY_601cc56abd30date__d_year, DBI16Type* KEY_601cc56abd30supplier__s_nation_encoded, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI32Type* lineorder__lo_custkey, DBI32Type* lineorder__lo_orderdate, DBDecimalType* lineorder__lo_revenue, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_601cc56f72a0 = 0;
auto reg_lineorder__lo_custkey = lineorder__lo_custkey[tid];

KEY_601cc56f72a0 |= reg_lineorder__lo_custkey;
//Probe Hash table
HT_601cc56f72a0.for_each(KEY_601cc56f72a0, [&] __device__ (auto const SLOT_601cc56f72a0) {
auto const [slot_first601cc56f72a0, slot_second601cc56f72a0] = SLOT_601cc56f72a0;
if (!(true)) return;
uint64_t KEY_601cc56f9a90 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_601cc56f9a90 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_601cc56f9a90.for_each(KEY_601cc56f9a90, [&] __device__ (auto const SLOT_601cc56f9a90) {
auto const [slot_first601cc56f9a90, slot_second601cc56f9a90] = SLOT_601cc56f9a90;
if (!(true)) return;
uint64_t KEY_601cc56dd2e0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_601cc56dd2e0 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_601cc56dd2e0.for_each(KEY_601cc56dd2e0, [&] __device__ (auto const SLOT_601cc56dd2e0) {
auto const [slot_first601cc56dd2e0, slot_second601cc56dd2e0] = SLOT_601cc56dd2e0;
if (!(true)) return;
uint64_t KEY_601cc56abd30 = 0;
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[BUF_601cc56f72a0[slot_second601cc56f72a0 * 1 + 0]];

KEY_601cc56abd30 |= reg_customer__c_nation_encoded;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[BUF_601cc56f9a90[slot_second601cc56f9a90 * 1 + 0]];
KEY_601cc56abd30 <<= 16;
KEY_601cc56abd30 |= reg_supplier__s_nation_encoded;
auto reg_date__d_year = date__d_year[BUF_601cc56dd2e0[slot_second601cc56dd2e0 * 1 + 0]];
KEY_601cc56abd30 <<= 32;
KEY_601cc56abd30 |= reg_date__d_year;
//Aggregate in hashtable
auto buf_idx_601cc56abd30 = HT_601cc56abd30.find(KEY_601cc56abd30)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_601cc56abd30], reg_lineorder__lo_revenue);
KEY_601cc56abd30customer__c_nation_encoded[buf_idx_601cc56abd30] = reg_customer__c_nation_encoded;
KEY_601cc56abd30supplier__s_nation_encoded[buf_idx_601cc56abd30] = reg_supplier__s_nation_encoded;
KEY_601cc56abd30date__d_year[buf_idx_601cc56abd30] = reg_date__d_year;
});
});
});
}
__global__ void count_601cc57160c0(uint64_t* COUNT601cc5687360, size_t COUNT601cc56abd30) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT601cc56abd30) return;
//Materialize count
atomicAdd((int*)COUNT601cc5687360, 1);
}
__global__ void main_601cc57160c0(size_t COUNT601cc56abd30, DBDecimalType* MAT601cc5687360aggr0__tmp_attr0, DBI16Type* MAT601cc5687360customer__c_nation_encoded, DBI32Type* MAT601cc5687360date__d_year, DBI16Type* MAT601cc5687360supplier__s_nation_encoded, uint64_t* MAT_IDX601cc5687360, DBDecimalType* aggr0__tmp_attr0, DBI16Type* customer__c_nation_encoded, DBI32Type* date__d_year, DBI16Type* supplier__s_nation_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT601cc56abd30) return;
//Materialize buffers
auto mat_idx601cc5687360 = atomicAdd((int*)MAT_IDX601cc5687360, 1);
auto reg_customer__c_nation_encoded = customer__c_nation_encoded[tid];
MAT601cc5687360customer__c_nation_encoded[mat_idx601cc5687360] = reg_customer__c_nation_encoded;
auto reg_supplier__s_nation_encoded = supplier__s_nation_encoded[tid];
MAT601cc5687360supplier__s_nation_encoded[mat_idx601cc5687360] = reg_supplier__s_nation_encoded;
auto reg_date__d_year = date__d_year[tid];
MAT601cc5687360date__d_year[mat_idx601cc5687360] = reg_date__d_year;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT601cc5687360aggr0__tmp_attr0[mat_idx601cc5687360] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT601cc56f72a0;
hipMalloc(&d_COUNT601cc56f72a0, sizeof(uint64_t));
hipMemset(d_COUNT601cc56f72a0, 0, sizeof(uint64_t));
count_601cc56dc0f0<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT601cc56f72a0, d_customer__c_region, customer_size);
uint64_t COUNT601cc56f72a0;
hipMemcpy(&COUNT601cc56f72a0, d_COUNT601cc56f72a0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_601cc56f72a0;
hipMalloc(&d_BUF_IDX_601cc56f72a0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_601cc56f72a0, 0, sizeof(uint64_t));
uint64_t* d_BUF_601cc56f72a0;
hipMalloc(&d_BUF_601cc56f72a0, sizeof(uint64_t) * COUNT601cc56f72a0 * 1);
auto d_HT_601cc56f72a0 = cuco::experimental::static_multimap{ (int)COUNT601cc56f72a0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_601cc56dc0f0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_601cc56f72a0, d_BUF_IDX_601cc56f72a0, d_HT_601cc56f72a0.ref(cuco::insert), d_customer__c_custkey, d_customer__c_region, customer_size);
//Materialize count
uint64_t* d_COUNT601cc56f9a90;
hipMalloc(&d_COUNT601cc56f9a90, sizeof(uint64_t));
hipMemset(d_COUNT601cc56f9a90, 0, sizeof(uint64_t));
count_601cc5702b80<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT601cc56f9a90, d_supplier__s_region, supplier_size);
uint64_t COUNT601cc56f9a90;
hipMemcpy(&COUNT601cc56f9a90, d_COUNT601cc56f9a90, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_601cc56f9a90;
hipMalloc(&d_BUF_IDX_601cc56f9a90, sizeof(uint64_t));
hipMemset(d_BUF_IDX_601cc56f9a90, 0, sizeof(uint64_t));
uint64_t* d_BUF_601cc56f9a90;
hipMalloc(&d_BUF_601cc56f9a90, sizeof(uint64_t) * COUNT601cc56f9a90 * 1);
auto d_HT_601cc56f9a90 = cuco::experimental::static_multimap{ (int)COUNT601cc56f9a90*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_601cc5702b80<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_601cc56f9a90, d_BUF_IDX_601cc56f9a90, d_HT_601cc56f9a90.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT601cc56dd2e0;
hipMalloc(&d_COUNT601cc56dd2e0, sizeof(uint64_t));
hipMemset(d_COUNT601cc56dd2e0, 0, sizeof(uint64_t));
count_601cc57089f0<<<std::ceil((float)date_size/32.), 32>>>(d_COUNT601cc56dd2e0, d_date__d_year, date_size);
uint64_t COUNT601cc56dd2e0;
hipMemcpy(&COUNT601cc56dd2e0, d_COUNT601cc56dd2e0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_601cc56dd2e0;
hipMalloc(&d_BUF_IDX_601cc56dd2e0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_601cc56dd2e0, 0, sizeof(uint64_t));
uint64_t* d_BUF_601cc56dd2e0;
hipMalloc(&d_BUF_601cc56dd2e0, sizeof(uint64_t) * COUNT601cc56dd2e0 * 1);
auto d_HT_601cc56dd2e0 = cuco::experimental::static_multimap{ (int)COUNT601cc56dd2e0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_601cc57089f0<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_601cc56dd2e0, d_BUF_IDX_601cc56dd2e0, d_HT_601cc56dd2e0.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Create aggregation hash table
auto d_HT_601cc56abd30 = cuco::static_map{ (int)144285*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_601cc56dc690<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_601cc56dd2e0, d_BUF_601cc56f72a0, d_BUF_601cc56f9a90, d_HT_601cc56abd30.ref(cuco::insert), d_HT_601cc56dd2e0.ref(cuco::for_each), d_HT_601cc56f72a0.ref(cuco::for_each), d_HT_601cc56f9a90.ref(cuco::for_each), d_customer__c_nation_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_nation_encoded);
size_t COUNT601cc56abd30 = d_HT_601cc56abd30.size();
thrust::device_vector<int64_t> keys_601cc56abd30(COUNT601cc56abd30), vals_601cc56abd30(COUNT601cc56abd30);
d_HT_601cc56abd30.retrieve_all(keys_601cc56abd30.begin(), vals_601cc56abd30.begin());
d_HT_601cc56abd30.clear();
int64_t* raw_keys601cc56abd30 = thrust::raw_pointer_cast(keys_601cc56abd30.data());
insertKeys<<<std::ceil((float)COUNT601cc56abd30/32.), 32>>>(raw_keys601cc56abd30, d_HT_601cc56abd30.ref(cuco::insert), COUNT601cc56abd30);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT601cc56abd30);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT601cc56abd30);
DBI16Type* d_KEY_601cc56abd30customer__c_nation_encoded;
hipMalloc(&d_KEY_601cc56abd30customer__c_nation_encoded, sizeof(DBI16Type) * COUNT601cc56abd30);
hipMemset(d_KEY_601cc56abd30customer__c_nation_encoded, 0, sizeof(DBI16Type) * COUNT601cc56abd30);
DBI16Type* d_KEY_601cc56abd30supplier__s_nation_encoded;
hipMalloc(&d_KEY_601cc56abd30supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT601cc56abd30);
hipMemset(d_KEY_601cc56abd30supplier__s_nation_encoded, 0, sizeof(DBI16Type) * COUNT601cc56abd30);
DBI32Type* d_KEY_601cc56abd30date__d_year;
hipMalloc(&d_KEY_601cc56abd30date__d_year, sizeof(DBI32Type) * COUNT601cc56abd30);
hipMemset(d_KEY_601cc56abd30date__d_year, 0, sizeof(DBI32Type) * COUNT601cc56abd30);
main_601cc56dc690<<<std::ceil((float)lineorder_size/256.), 256>>>(d_BUF_601cc56dd2e0, d_BUF_601cc56f72a0, d_BUF_601cc56f9a90, d_HT_601cc56abd30.ref(cuco::find), d_HT_601cc56dd2e0.ref(cuco::for_each), d_HT_601cc56f72a0.ref(cuco::for_each), d_HT_601cc56f9a90.ref(cuco::for_each), d_KEY_601cc56abd30customer__c_nation_encoded, d_KEY_601cc56abd30date__d_year, d_KEY_601cc56abd30supplier__s_nation_encoded, d_aggr0__tmp_attr0, d_customer__c_nation_encoded, d_date__d_year, d_lineorder__lo_custkey, d_lineorder__lo_orderdate, d_lineorder__lo_revenue, d_lineorder__lo_suppkey, lineorder_size, d_supplier__s_nation_encoded);
//Materialize count
uint64_t* d_COUNT601cc5687360;
hipMalloc(&d_COUNT601cc5687360, sizeof(uint64_t));
hipMemset(d_COUNT601cc5687360, 0, sizeof(uint64_t));
count_601cc57160c0<<<std::ceil((float)COUNT601cc56abd30/32.), 32>>>(d_COUNT601cc5687360, COUNT601cc56abd30);
uint64_t COUNT601cc5687360;
hipMemcpy(&COUNT601cc5687360, d_COUNT601cc5687360, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX601cc5687360;
hipMalloc(&d_MAT_IDX601cc5687360, sizeof(uint64_t));
hipMemset(d_MAT_IDX601cc5687360, 0, sizeof(uint64_t));
auto MAT601cc5687360customer__c_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT601cc5687360);
DBI16Type* d_MAT601cc5687360customer__c_nation_encoded;
hipMalloc(&d_MAT601cc5687360customer__c_nation_encoded, sizeof(DBI16Type) * COUNT601cc5687360);
auto MAT601cc5687360supplier__s_nation_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT601cc5687360);
DBI16Type* d_MAT601cc5687360supplier__s_nation_encoded;
hipMalloc(&d_MAT601cc5687360supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT601cc5687360);
auto MAT601cc5687360date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT601cc5687360);
DBI32Type* d_MAT601cc5687360date__d_year;
hipMalloc(&d_MAT601cc5687360date__d_year, sizeof(DBI32Type) * COUNT601cc5687360);
auto MAT601cc5687360aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT601cc5687360);
DBDecimalType* d_MAT601cc5687360aggr0__tmp_attr0;
hipMalloc(&d_MAT601cc5687360aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT601cc5687360);
main_601cc57160c0<<<std::ceil((float)COUNT601cc56abd30/32.), 32>>>(COUNT601cc56abd30, d_MAT601cc5687360aggr0__tmp_attr0, d_MAT601cc5687360customer__c_nation_encoded, d_MAT601cc5687360date__d_year, d_MAT601cc5687360supplier__s_nation_encoded, d_MAT_IDX601cc5687360, d_aggr0__tmp_attr0, d_KEY_601cc56abd30customer__c_nation_encoded, d_KEY_601cc56abd30date__d_year, d_KEY_601cc56abd30supplier__s_nation_encoded);
hipMemcpy(MAT601cc5687360customer__c_nation_encoded, d_MAT601cc5687360customer__c_nation_encoded, sizeof(DBI16Type) * COUNT601cc5687360, hipMemcpyDeviceToHost);
hipMemcpy(MAT601cc5687360supplier__s_nation_encoded, d_MAT601cc5687360supplier__s_nation_encoded, sizeof(DBI16Type) * COUNT601cc5687360, hipMemcpyDeviceToHost);
hipMemcpy(MAT601cc5687360date__d_year, d_MAT601cc5687360date__d_year, sizeof(DBI32Type) * COUNT601cc5687360, hipMemcpyDeviceToHost);
hipMemcpy(MAT601cc5687360aggr0__tmp_attr0, d_MAT601cc5687360aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT601cc5687360, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT601cc5687360; i++) { std::cout << customer__c_nation_map[MAT601cc5687360customer__c_nation_encoded[i]] << "\t";
std::cout << supplier__s_nation_map[MAT601cc5687360supplier__s_nation_encoded[i]] << "\t";
std::cout << MAT601cc5687360date__d_year[i] << "\t";
std::cout << MAT601cc5687360aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_601cc56f72a0);
hipFree(d_BUF_IDX_601cc56f72a0);
hipFree(d_COUNT601cc56f72a0);
hipFree(d_BUF_601cc56f9a90);
hipFree(d_BUF_IDX_601cc56f9a90);
hipFree(d_COUNT601cc56f9a90);
hipFree(d_BUF_601cc56dd2e0);
hipFree(d_BUF_IDX_601cc56dd2e0);
hipFree(d_COUNT601cc56dd2e0);
hipFree(d_KEY_601cc56abd30customer__c_nation_encoded);
hipFree(d_KEY_601cc56abd30date__d_year);
hipFree(d_KEY_601cc56abd30supplier__s_nation_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT601cc5687360);
hipFree(d_MAT601cc5687360aggr0__tmp_attr0);
hipFree(d_MAT601cc5687360customer__c_nation_encoded);
hipFree(d_MAT601cc5687360date__d_year);
hipFree(d_MAT601cc5687360supplier__s_nation_encoded);
hipFree(d_MAT_IDX601cc5687360);
free(MAT601cc5687360aggr0__tmp_attr0);
free(MAT601cc5687360customer__c_nation_encoded);
free(MAT601cc5687360date__d_year);
free(MAT601cc5687360supplier__s_nation_encoded);
}