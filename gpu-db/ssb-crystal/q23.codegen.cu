#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5a87f76ec180(uint64_t* COUNT5a87f76e09c0, DBStringType* part__p_brand1, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_brand1 = part__p_brand1[tid];
if (!(evaluatePredicate(reg_part__p_brand1, "MFGR#2239", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5a87f76e09c0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5a87f76ec180(uint64_t* BUF_5a87f76e09c0, uint64_t* BUF_IDX_5a87f76e09c0, HASHTABLE_INSERT HT_5a87f76e09c0, DBStringType* part__p_brand1, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_brand1 = part__p_brand1[tid];
if (!(evaluatePredicate(reg_part__p_brand1, "MFGR#2239", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5a87f76e09c0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_5a87f76e09c0 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_5a87f76e09c0 = atomicAdd((int*)BUF_IDX_5a87f76e09c0, 1);
HT_5a87f76e09c0.insert(cuco::pair{KEY_5a87f76e09c0, buf_idx_5a87f76e09c0});
BUF_5a87f76e09c0[buf_idx_5a87f76e09c0 * 1 + 0] = tid;
}
__global__ void count_5a87f76eeba0(uint64_t* COUNT5a87f76e0660, DBStringType* supplier__s_region, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "EUROPE", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5a87f76e0660, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5a87f76eeba0(uint64_t* BUF_5a87f76e0660, uint64_t* BUF_IDX_5a87f76e0660, HASHTABLE_INSERT HT_5a87f76e0660, DBStringType* supplier__s_region, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
auto reg_supplier__s_region = supplier__s_region[tid];
if (!(evaluatePredicate(reg_supplier__s_region, "EUROPE", Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5a87f76e0660 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5a87f76e0660 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5a87f76e0660 = atomicAdd((int*)BUF_IDX_5a87f76e0660, 1);
HT_5a87f76e0660.insert(cuco::pair{KEY_5a87f76e0660, buf_idx_5a87f76e0660});
BUF_5a87f76e0660[buf_idx_5a87f76e0660 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5a87f76c49a0(uint64_t* BUF_5a87f76e0660, uint64_t* BUF_5a87f76e09c0, uint64_t* COUNT5a87f76db5d0, HASHTABLE_PROBE HT_5a87f76e0660, HASHTABLE_PROBE HT_5a87f76e09c0, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5a87f76e09c0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5a87f76e09c0 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5a87f76e09c0.for_each(KEY_5a87f76e09c0, [&] __device__ (auto const SLOT_5a87f76e09c0) {

auto const [slot_first5a87f76e09c0, slot_second5a87f76e09c0] = SLOT_5a87f76e09c0;
if (!(true)) return;
uint64_t KEY_5a87f76e0660 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5a87f76e0660 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5a87f76e0660.for_each(KEY_5a87f76e0660, [&] __device__ (auto const SLOT_5a87f76e0660) {

auto const [slot_first5a87f76e0660, slot_second5a87f76e0660] = SLOT_5a87f76e0660;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5a87f76db5d0, 1);
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_5a87f76c49a0(uint64_t* BUF_5a87f76db5d0, uint64_t* BUF_5a87f76e0660, uint64_t* BUF_5a87f76e09c0, uint64_t* BUF_IDX_5a87f76db5d0, HASHTABLE_INSERT HT_5a87f76db5d0, HASHTABLE_PROBE HT_5a87f76e0660, HASHTABLE_PROBE HT_5a87f76e09c0, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_partkey, DBI32Type* lineorder__lo_suppkey, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5a87f76e09c0 = 0;
auto reg_lineorder__lo_partkey = lineorder__lo_partkey[tid];

KEY_5a87f76e09c0 |= reg_lineorder__lo_partkey;
//Probe Hash table
HT_5a87f76e09c0.for_each(KEY_5a87f76e09c0, [&] __device__ (auto const SLOT_5a87f76e09c0) {
auto const [slot_first5a87f76e09c0, slot_second5a87f76e09c0] = SLOT_5a87f76e09c0;
if (!(true)) return;
uint64_t KEY_5a87f76e0660 = 0;
auto reg_lineorder__lo_suppkey = lineorder__lo_suppkey[tid];

KEY_5a87f76e0660 |= reg_lineorder__lo_suppkey;
//Probe Hash table
HT_5a87f76e0660.for_each(KEY_5a87f76e0660, [&] __device__ (auto const SLOT_5a87f76e0660) {
auto const [slot_first5a87f76e0660, slot_second5a87f76e0660] = SLOT_5a87f76e0660;
if (!(true)) return;
uint64_t KEY_5a87f76db5d0 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5a87f76db5d0 |= reg_lineorder__lo_orderdate;
// Insert hash table kernel;
auto buf_idx_5a87f76db5d0 = atomicAdd((int*)BUF_IDX_5a87f76db5d0, 1);
HT_5a87f76db5d0.insert(cuco::pair{KEY_5a87f76db5d0, buf_idx_5a87f76db5d0});
BUF_5a87f76db5d0[buf_idx_5a87f76db5d0 * 3 + 0] = tid;
BUF_5a87f76db5d0[buf_idx_5a87f76db5d0 * 3 + 1] = BUF_5a87f76e09c0[slot_second5a87f76e09c0 * 1 + 0];
BUF_5a87f76db5d0[buf_idx_5a87f76db5d0 * 3 + 2] = BUF_5a87f76e0660[slot_second5a87f76e0660 * 1 + 0];
});
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5a87f76c4f70(uint64_t* BUF_5a87f76db5d0, HASHTABLE_INSERT HT_5a87f7695350, HASHTABLE_PROBE HT_5a87f76db5d0, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5a87f76db5d0 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5a87f76db5d0 |= reg_date__d_datekey;
//Probe Hash table
HT_5a87f76db5d0.for_each(KEY_5a87f76db5d0, [&] __device__ (auto const SLOT_5a87f76db5d0) {

auto const [slot_first5a87f76db5d0, slot_second5a87f76db5d0] = SLOT_5a87f76db5d0;
if (!(true)) return;
uint64_t KEY_5a87f7695350 = 0;
auto reg_date__d_year = date__d_year[tid];

KEY_5a87f7695350 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_5a87f76db5d0[slot_second5a87f76db5d0 * 3 + 1]];
KEY_5a87f7695350 <<= 16;
KEY_5a87f7695350 |= reg_part__p_brand1_encoded;
//Create aggregation hash table
HT_5a87f7695350.insert(cuco::pair{KEY_5a87f7695350, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5a87f76c4f70(uint64_t* BUF_5a87f76db5d0, HASHTABLE_FIND HT_5a87f7695350, HASHTABLE_PROBE HT_5a87f76db5d0, DBI32Type* KEY_5a87f7695350date__d_year, DBI16Type* KEY_5a87f7695350part__p_brand1_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size, DBDecimalType* lineorder__lo_revenue, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5a87f76db5d0 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5a87f76db5d0 |= reg_date__d_datekey;
//Probe Hash table
HT_5a87f76db5d0.for_each(KEY_5a87f76db5d0, [&] __device__ (auto const SLOT_5a87f76db5d0) {
auto const [slot_first5a87f76db5d0, slot_second5a87f76db5d0] = SLOT_5a87f76db5d0;
if (!(true)) return;
uint64_t KEY_5a87f7695350 = 0;
auto reg_date__d_year = date__d_year[tid];

KEY_5a87f7695350 |= reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[BUF_5a87f76db5d0[slot_second5a87f76db5d0 * 3 + 1]];
KEY_5a87f7695350 <<= 16;
KEY_5a87f7695350 |= reg_part__p_brand1_encoded;
//Aggregate in hashtable
auto buf_idx_5a87f7695350 = HT_5a87f7695350.find(KEY_5a87f7695350)->second;
auto reg_lineorder__lo_revenue = lineorder__lo_revenue[BUF_5a87f76db5d0[slot_second5a87f76db5d0 * 3 + 0]];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5a87f7695350], reg_lineorder__lo_revenue);
KEY_5a87f7695350date__d_year[buf_idx_5a87f7695350] = reg_date__d_year;
KEY_5a87f7695350part__p_brand1_encoded[buf_idx_5a87f7695350] = reg_part__p_brand1_encoded;
});
}
__global__ void count_5a87f76ff370(size_t COUNT5a87f7695350, uint64_t* COUNT5a87f76a8470) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5a87f7695350) return;
//Materialize count
atomicAdd((int*)COUNT5a87f76a8470, 1);
}
__global__ void main_5a87f76ff370(size_t COUNT5a87f7695350, DBDecimalType* MAT5a87f76a8470aggr0__tmp_attr0, DBI32Type* MAT5a87f76a8470date__d_year, DBI16Type* MAT5a87f76a8470part__p_brand1_encoded, uint64_t* MAT_IDX5a87f76a8470, DBDecimalType* aggr0__tmp_attr0, DBI32Type* date__d_year, DBI16Type* part__p_brand1_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5a87f7695350) return;
//Materialize buffers
auto mat_idx5a87f76a8470 = atomicAdd((int*)MAT_IDX5a87f76a8470, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5a87f76a8470aggr0__tmp_attr0[mat_idx5a87f76a8470] = reg_aggr0__tmp_attr0;
auto reg_date__d_year = date__d_year[tid];
MAT5a87f76a8470date__d_year[mat_idx5a87f76a8470] = reg_date__d_year;
auto reg_part__p_brand1_encoded = part__p_brand1_encoded[tid];
MAT5a87f76a8470part__p_brand1_encoded[mat_idx5a87f76a8470] = reg_part__p_brand1_encoded;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT5a87f76e09c0;
hipMalloc(&d_COUNT5a87f76e09c0, sizeof(uint64_t));
hipMemset(d_COUNT5a87f76e09c0, 0, sizeof(uint64_t));
count_5a87f76ec180<<<std::ceil((float)part_size/32.), 32>>>(d_COUNT5a87f76e09c0, d_part__p_brand1, part_size);
uint64_t COUNT5a87f76e09c0;
hipMemcpy(&COUNT5a87f76e09c0, d_COUNT5a87f76e09c0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5a87f76e09c0;
hipMalloc(&d_BUF_IDX_5a87f76e09c0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5a87f76e09c0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5a87f76e09c0;
hipMalloc(&d_BUF_5a87f76e09c0, sizeof(uint64_t) * COUNT5a87f76e09c0 * 1);
auto d_HT_5a87f76e09c0 = cuco::experimental::static_multimap{ (int)COUNT5a87f76e09c0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5a87f76ec180<<<std::ceil((float)part_size/32.), 32>>>(d_BUF_5a87f76e09c0, d_BUF_IDX_5a87f76e09c0, d_HT_5a87f76e09c0.ref(cuco::insert), d_part__p_brand1, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT5a87f76e0660;
hipMalloc(&d_COUNT5a87f76e0660, sizeof(uint64_t));
hipMemset(d_COUNT5a87f76e0660, 0, sizeof(uint64_t));
count_5a87f76eeba0<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT5a87f76e0660, d_supplier__s_region, supplier_size);
uint64_t COUNT5a87f76e0660;
hipMemcpy(&COUNT5a87f76e0660, d_COUNT5a87f76e0660, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5a87f76e0660;
hipMalloc(&d_BUF_IDX_5a87f76e0660, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5a87f76e0660, 0, sizeof(uint64_t));
uint64_t* d_BUF_5a87f76e0660;
hipMalloc(&d_BUF_5a87f76e0660, sizeof(uint64_t) * COUNT5a87f76e0660 * 1);
auto d_HT_5a87f76e0660 = cuco::experimental::static_multimap{ (int)COUNT5a87f76e0660*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5a87f76eeba0<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_5a87f76e0660, d_BUF_IDX_5a87f76e0660, d_HT_5a87f76e0660.ref(cuco::insert), d_supplier__s_region, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5a87f76db5d0;
hipMalloc(&d_COUNT5a87f76db5d0, sizeof(uint64_t));
hipMemset(d_COUNT5a87f76db5d0, 0, sizeof(uint64_t));
count_5a87f76c49a0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5a87f76e0660, d_BUF_5a87f76e09c0, d_COUNT5a87f76db5d0, d_HT_5a87f76e0660.ref(cuco::for_each), d_HT_5a87f76e09c0.ref(cuco::for_each), d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
uint64_t COUNT5a87f76db5d0;
hipMemcpy(&COUNT5a87f76db5d0, d_COUNT5a87f76db5d0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5a87f76db5d0;
hipMalloc(&d_BUF_IDX_5a87f76db5d0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5a87f76db5d0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5a87f76db5d0;
hipMalloc(&d_BUF_5a87f76db5d0, sizeof(uint64_t) * COUNT5a87f76db5d0 * 3);
auto d_HT_5a87f76db5d0 = cuco::experimental::static_multimap{ (int)COUNT5a87f76db5d0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5a87f76c49a0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_5a87f76db5d0, d_BUF_5a87f76e0660, d_BUF_5a87f76e09c0, d_BUF_IDX_5a87f76db5d0, d_HT_5a87f76db5d0.ref(cuco::insert), d_HT_5a87f76e0660.ref(cuco::for_each), d_HT_5a87f76e09c0.ref(cuco::for_each), d_lineorder__lo_orderdate, d_lineorder__lo_partkey, d_lineorder__lo_suppkey, lineorder_size);
//Create aggregation hash table
auto d_HT_5a87f7695350 = cuco::static_map{ (int)1208*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5a87f76c4f70<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_5a87f76db5d0, d_HT_5a87f7695350.ref(cuco::insert), d_HT_5a87f76db5d0.ref(cuco::for_each), d_date__d_datekey, d_date__d_year, date_size, d_part__p_brand1_encoded);
size_t COUNT5a87f7695350 = d_HT_5a87f7695350.size();
thrust::device_vector<int64_t> keys_5a87f7695350(COUNT5a87f7695350), vals_5a87f7695350(COUNT5a87f7695350);
d_HT_5a87f7695350.retrieve_all(keys_5a87f7695350.begin(), vals_5a87f7695350.begin());
d_HT_5a87f7695350.clear();
int64_t* raw_keys5a87f7695350 = thrust::raw_pointer_cast(keys_5a87f7695350.data());
insertKeys<<<std::ceil((float)COUNT5a87f7695350/32.), 32>>>(raw_keys5a87f7695350, d_HT_5a87f7695350.ref(cuco::insert), COUNT5a87f7695350);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5a87f7695350);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5a87f7695350);
DBI32Type* d_KEY_5a87f7695350date__d_year;
hipMalloc(&d_KEY_5a87f7695350date__d_year, sizeof(DBI32Type) * COUNT5a87f7695350);
hipMemset(d_KEY_5a87f7695350date__d_year, 0, sizeof(DBI32Type) * COUNT5a87f7695350);
DBI16Type* d_KEY_5a87f7695350part__p_brand1_encoded;
hipMalloc(&d_KEY_5a87f7695350part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5a87f7695350);
hipMemset(d_KEY_5a87f7695350part__p_brand1_encoded, 0, sizeof(DBI16Type) * COUNT5a87f7695350);
main_5a87f76c4f70<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_5a87f76db5d0, d_HT_5a87f7695350.ref(cuco::find), d_HT_5a87f76db5d0.ref(cuco::for_each), d_KEY_5a87f7695350date__d_year, d_KEY_5a87f7695350part__p_brand1_encoded, d_aggr0__tmp_attr0, d_date__d_datekey, d_date__d_year, date_size, d_lineorder__lo_revenue, d_part__p_brand1_encoded);
//Materialize count
uint64_t* d_COUNT5a87f76a8470;
hipMalloc(&d_COUNT5a87f76a8470, sizeof(uint64_t));
hipMemset(d_COUNT5a87f76a8470, 0, sizeof(uint64_t));
count_5a87f76ff370<<<std::ceil((float)COUNT5a87f7695350/32.), 32>>>(COUNT5a87f7695350, d_COUNT5a87f76a8470);
uint64_t COUNT5a87f76a8470;
hipMemcpy(&COUNT5a87f76a8470, d_COUNT5a87f76a8470, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5a87f76a8470;
hipMalloc(&d_MAT_IDX5a87f76a8470, sizeof(uint64_t));
hipMemset(d_MAT_IDX5a87f76a8470, 0, sizeof(uint64_t));
auto MAT5a87f76a8470aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5a87f76a8470);
DBDecimalType* d_MAT5a87f76a8470aggr0__tmp_attr0;
hipMalloc(&d_MAT5a87f76a8470aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5a87f76a8470);
auto MAT5a87f76a8470date__d_year = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5a87f76a8470);
DBI32Type* d_MAT5a87f76a8470date__d_year;
hipMalloc(&d_MAT5a87f76a8470date__d_year, sizeof(DBI32Type) * COUNT5a87f76a8470);
auto MAT5a87f76a8470part__p_brand1_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT5a87f76a8470);
DBI16Type* d_MAT5a87f76a8470part__p_brand1_encoded;
hipMalloc(&d_MAT5a87f76a8470part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5a87f76a8470);
main_5a87f76ff370<<<std::ceil((float)COUNT5a87f7695350/32.), 32>>>(COUNT5a87f7695350, d_MAT5a87f76a8470aggr0__tmp_attr0, d_MAT5a87f76a8470date__d_year, d_MAT5a87f76a8470part__p_brand1_encoded, d_MAT_IDX5a87f76a8470, d_aggr0__tmp_attr0, d_KEY_5a87f7695350date__d_year, d_KEY_5a87f7695350part__p_brand1_encoded);
hipMemcpy(MAT5a87f76a8470aggr0__tmp_attr0, d_MAT5a87f76a8470aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5a87f76a8470, hipMemcpyDeviceToHost);
hipMemcpy(MAT5a87f76a8470date__d_year, d_MAT5a87f76a8470date__d_year, sizeof(DBI32Type) * COUNT5a87f76a8470, hipMemcpyDeviceToHost);
hipMemcpy(MAT5a87f76a8470part__p_brand1_encoded, d_MAT5a87f76a8470part__p_brand1_encoded, sizeof(DBI16Type) * COUNT5a87f76a8470, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5a87f76a8470; i++) { std::cout << MAT5a87f76a8470aggr0__tmp_attr0[i] << "\t";
std::cout << MAT5a87f76a8470date__d_year[i] << "\t";
std::cout << part__p_brand1_map[MAT5a87f76a8470part__p_brand1_encoded[i]] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_5a87f76e09c0);
hipFree(d_BUF_IDX_5a87f76e09c0);
hipFree(d_COUNT5a87f76e09c0);
hipFree(d_BUF_5a87f76e0660);
hipFree(d_BUF_IDX_5a87f76e0660);
hipFree(d_COUNT5a87f76e0660);
hipFree(d_BUF_5a87f76db5d0);
hipFree(d_BUF_IDX_5a87f76db5d0);
hipFree(d_COUNT5a87f76db5d0);
hipFree(d_KEY_5a87f7695350date__d_year);
hipFree(d_KEY_5a87f7695350part__p_brand1_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5a87f76a8470);
hipFree(d_MAT5a87f76a8470aggr0__tmp_attr0);
hipFree(d_MAT5a87f76a8470date__d_year);
hipFree(d_MAT5a87f76a8470part__p_brand1_encoded);
hipFree(d_MAT_IDX5a87f76a8470);
free(MAT5a87f76a8470aggr0__tmp_attr0);
free(MAT5a87f76a8470date__d_year);
free(MAT5a87f76a8470part__p_brand1_encoded);
}