#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5e8b6f444c30(uint64_t* COUNT5e8b6f45c390, DBI32Type* date__d_weeknuminyear, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_weeknuminyear = date__d_weeknuminyear[tid];
if (!(evaluatePredicate(reg_date__d_weeknuminyear, 6, Predicate::eq))) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1994, Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT5e8b6f45c390, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5e8b6f444c30(uint64_t* BUF_5e8b6f45c390, uint64_t* BUF_IDX_5e8b6f45c390, HASHTABLE_INSERT HT_5e8b6f45c390, DBI32Type* date__d_datekey, DBI32Type* date__d_weeknuminyear, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_weeknuminyear = date__d_weeknuminyear[tid];
if (!(evaluatePredicate(reg_date__d_weeknuminyear, 6, Predicate::eq))) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1994, Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e8b6f45c390 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_5e8b6f45c390 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_5e8b6f45c390 = atomicAdd((int*)BUF_IDX_5e8b6f45c390, 1);
HT_5e8b6f45c390.insert(cuco::pair{KEY_5e8b6f45c390, buf_idx_5e8b6f45c390});
BUF_5e8b6f45c390[buf_idx_5e8b6f45c390 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5e8b6f444740(uint64_t* BUF_5e8b6f45c390, HASHTABLE_INSERT HT_5e8b6f4159e0, HASHTABLE_PROBE HT_5e8b6f45c390, DBI32Type* lineorder__lo_discount, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_discount = lineorder__lo_discount[tid];
if (!(evaluatePredicate(reg_lineorder__lo_discount, 5, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount, 7, Predicate::lte))) return;
auto reg_lineorder__lo_quantity = lineorder__lo_quantity[tid];
if (!(evaluatePredicate(reg_lineorder__lo_quantity, 26, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_quantity, 35, Predicate::lte))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_5e8b6f45c390 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_5e8b6f45c390 |= reg_lineorder__lo_orderdate;
//Probe Hash table
HT_5e8b6f45c390.for_each(KEY_5e8b6f45c390, [&] __device__ (auto const SLOT_5e8b6f45c390) {

auto const [slot_first5e8b6f45c390, slot_second5e8b6f45c390] = SLOT_5e8b6f45c390;
if (!(true)) return;
uint64_t KEY_5e8b6f4159e0 = 0;
//Create aggregation hash table
HT_5e8b6f4159e0.insert(cuco::pair{KEY_5e8b6f4159e0, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5e8b6f444740(uint64_t* BUF_5e8b6f45c390, HASHTABLE_FIND HT_5e8b6f4159e0, HASHTABLE_PROBE HT_5e8b6f45c390, DBDecimalType* aggr0__tmp_attr0, DBI32Type* lineorder__lo_discount, DBDecimalType* lineorder__lo_extendedprice, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_discount = lineorder__lo_discount[tid];
__shared__ DBDecimalType ag;
ag = 0.;
__syncthreads();
if ((evaluatePredicate(reg_lineorder__lo_discount, 5, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount, 7, Predicate::lte))) {

    auto reg_lineorder__lo_quantity = lineorder__lo_quantity[tid];
    if ((evaluatePredicate(reg_lineorder__lo_quantity, 26, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_quantity, 35, Predicate::lte))) {

        if (!(!(false))) return;
        if (!(!(false))) return;
        if (!(!(false))) return;
        if (!(!(false))) return;
        if (!(!(false))) return;
        if (!(!(false))) return;
        uint64_t KEY_5e8b6f45c390 = 0;
        auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];
        
        KEY_5e8b6f45c390 |= reg_lineorder__lo_orderdate;
        //Probe Hash table
        HT_5e8b6f45c390.for_each(KEY_5e8b6f45c390, [&] __device__ (auto const SLOT_5e8b6f45c390) {
            auto const [slot_first5e8b6f45c390, slot_second5e8b6f45c390] = SLOT_5e8b6f45c390;
            if (!(true)) return;
            uint64_t KEY_5e8b6f4159e0 = 0;
            //Aggregate in hashtable
            auto buf_idx_5e8b6f4159e0 = HT_5e8b6f4159e0.find(KEY_5e8b6f4159e0)->second;
            auto reg_lineorder__lo_extendedprice = lineorder__lo_extendedprice[tid];
            auto reg_map0__tmp_attr1 = (reg_lineorder__lo_extendedprice) * ((DBDecimalType)(reg_lineorder__lo_discount));
            aggregate_sum(&ag, reg_map0__tmp_attr1);
            // aggregate_sum(&aggr0__tmp_attr0[buf_idx_5e8b6f4159e0], reg_map0__tmp_attr1);
        });
    }
}
__syncthreads();
if (threadIdx.x == 0) {
    aggregate_sum(&aggr0__tmp_attr0[0], ag);
}
}
__global__ void count_5e8b6f46e6a0(size_t COUNT5e8b6f4159e0, uint64_t* COUNT5e8b6f427d70) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5e8b6f4159e0) return;
//Materialize count
atomicAdd((int*)COUNT5e8b6f427d70, 1);
}
__global__ void main_5e8b6f46e6a0(size_t COUNT5e8b6f4159e0, DBDecimalType* MAT5e8b6f427d70aggr0__tmp_attr0, uint64_t* MAT_IDX5e8b6f427d70, DBDecimalType* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5e8b6f4159e0) return;
//Materialize buffers
auto mat_idx5e8b6f427d70 = atomicAdd((int*)MAT_IDX5e8b6f427d70, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5e8b6f427d70aggr0__tmp_attr0[mat_idx5e8b6f427d70] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT5e8b6f45c390;
hipMalloc(&d_COUNT5e8b6f45c390, sizeof(uint64_t));
hipMemset(d_COUNT5e8b6f45c390, 0, sizeof(uint64_t));
count_5e8b6f444c30<<<std::ceil((float)date_size/32.), 32>>>(d_COUNT5e8b6f45c390, d_date__d_weeknuminyear, d_date__d_year, date_size);
uint64_t COUNT5e8b6f45c390;
hipMemcpy(&COUNT5e8b6f45c390, d_COUNT5e8b6f45c390, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5e8b6f45c390;
hipMalloc(&d_BUF_IDX_5e8b6f45c390, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5e8b6f45c390, 0, sizeof(uint64_t));
uint64_t* d_BUF_5e8b6f45c390;
hipMalloc(&d_BUF_5e8b6f45c390, sizeof(uint64_t) * COUNT5e8b6f45c390 * 1);
auto d_HT_5e8b6f45c390 = cuco::experimental::static_multimap{ (int)COUNT5e8b6f45c390*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5e8b6f444c30<<<std::ceil((float)date_size/32.), 32>>>(d_BUF_5e8b6f45c390, d_BUF_IDX_5e8b6f45c390, d_HT_5e8b6f45c390.ref(cuco::insert), d_date__d_datekey, d_date__d_weeknuminyear, d_date__d_year, date_size);
//Create aggregation hash table
auto d_HT_5e8b6f4159e0 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5e8b6f444740<<<std::ceil((float)lineorder_size/32.), 32>>>(d_BUF_5e8b6f45c390, d_HT_5e8b6f4159e0.ref(cuco::insert), d_HT_5e8b6f45c390.ref(cuco::for_each), d_lineorder__lo_discount, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
size_t COUNT5e8b6f4159e0 = d_HT_5e8b6f4159e0.size();
thrust::device_vector<int64_t> keys_5e8b6f4159e0(COUNT5e8b6f4159e0), vals_5e8b6f4159e0(COUNT5e8b6f4159e0);
d_HT_5e8b6f4159e0.retrieve_all(keys_5e8b6f4159e0.begin(), vals_5e8b6f4159e0.begin());
d_HT_5e8b6f4159e0.clear();
int64_t* raw_keys5e8b6f4159e0 = thrust::raw_pointer_cast(keys_5e8b6f4159e0.data());
insertKeys<<<std::ceil((float)COUNT5e8b6f4159e0/32.), 32>>>(raw_keys5e8b6f4159e0, d_HT_5e8b6f4159e0.ref(cuco::insert), COUNT5e8b6f4159e0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5e8b6f4159e0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5e8b6f4159e0);
main_5e8b6f444740<<<std::ceil((float)lineorder_size/256.), 256>>>(d_BUF_5e8b6f45c390, d_HT_5e8b6f4159e0.ref(cuco::find), d_HT_5e8b6f45c390.ref(cuco::for_each), d_aggr0__tmp_attr0, d_lineorder__lo_discount, d_lineorder__lo_extendedprice, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
//Materialize count
uint64_t* d_COUNT5e8b6f427d70;
hipMalloc(&d_COUNT5e8b6f427d70, sizeof(uint64_t));
hipMemset(d_COUNT5e8b6f427d70, 0, sizeof(uint64_t));
count_5e8b6f46e6a0<<<std::ceil((float)COUNT5e8b6f4159e0/32.), 32>>>(COUNT5e8b6f4159e0, d_COUNT5e8b6f427d70);
uint64_t COUNT5e8b6f427d70;
hipMemcpy(&COUNT5e8b6f427d70, d_COUNT5e8b6f427d70, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5e8b6f427d70;
hipMalloc(&d_MAT_IDX5e8b6f427d70, sizeof(uint64_t));
hipMemset(d_MAT_IDX5e8b6f427d70, 0, sizeof(uint64_t));
auto MAT5e8b6f427d70aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5e8b6f427d70);
DBDecimalType* d_MAT5e8b6f427d70aggr0__tmp_attr0;
hipMalloc(&d_MAT5e8b6f427d70aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5e8b6f427d70);
main_5e8b6f46e6a0<<<std::ceil((float)COUNT5e8b6f4159e0/32.), 32>>>(COUNT5e8b6f4159e0, d_MAT5e8b6f427d70aggr0__tmp_attr0, d_MAT_IDX5e8b6f427d70, d_aggr0__tmp_attr0);
hipMemcpy(MAT5e8b6f427d70aggr0__tmp_attr0, d_MAT5e8b6f427d70aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5e8b6f427d70, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5e8b6f427d70; i++) { std::cout << MAT5e8b6f427d70aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_5e8b6f45c390);
hipFree(d_BUF_IDX_5e8b6f45c390);
hipFree(d_COUNT5e8b6f45c390);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT5e8b6f427d70);
hipFree(d_MAT5e8b6f427d70aggr0__tmp_attr0);
hipFree(d_MAT_IDX5e8b6f427d70);
free(MAT5e8b6f427d70aggr0__tmp_attr0);
}