#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_572ef7b75a90(uint64_t* COUNT572ef7b8ac20, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1993, Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
//Materialize count
atomicAdd((int*)COUNT572ef7b8ac20, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_572ef7b75a90(uint64_t* BUF_572ef7b8ac20, uint64_t* BUF_IDX_572ef7b8ac20, HASHTABLE_INSERT HT_572ef7b8ac20, DBI32Type* date__d_datekey, DBI32Type* date__d_year, size_t date_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= date_size) return;
auto reg_date__d_year = date__d_year[tid];
if (!(evaluatePredicate(reg_date__d_year, 1993, Predicate::eq))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_572ef7b8ac20 = 0;
auto reg_date__d_datekey = date__d_datekey[tid];

KEY_572ef7b8ac20 |= reg_date__d_datekey;
// Insert hash table kernel;
auto buf_idx_572ef7b8ac20 = atomicAdd((int*)BUF_IDX_572ef7b8ac20, 1);
HT_572ef7b8ac20.insert(cuco::pair{KEY_572ef7b8ac20, buf_idx_572ef7b8ac20});
BUF_572ef7b8ac20[buf_idx_572ef7b8ac20 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_572ef7b755c0(uint64_t* BUF_572ef7b8ac20, HASHTABLE_INSERT HT_572ef7b45c40, HASHTABLE_PROBE HT_572ef7b8ac20, DBI32Type* lineorder__lo_discount, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
auto reg_lineorder__lo_discount = lineorder__lo_discount[tid];
if (!(evaluatePredicate(reg_lineorder__lo_discount, 1, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount, 3, Predicate::lte))) return;
auto reg_lineorder__lo_quantity = lineorder__lo_quantity[tid];
if (!(evaluatePredicate(reg_lineorder__lo_quantity, 25, Predicate::lt))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
if (!(!(false))) return;
uint64_t KEY_572ef7b8ac20 = 0;
auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];

KEY_572ef7b8ac20 |= reg_lineorder__lo_orderdate;
//Probe Hash table
auto SLOT_572ef7b8ac20 = HT_572ef7b8ac20.find(KEY_572ef7b8ac20);
if (SLOT_572ef7b8ac20 == HT_572ef7b8ac20.end()) return;
if (!(true)) return;
uint64_t KEY_572ef7b45c40 = 0;
//Create aggregation hash table
HT_572ef7b45c40.insert(cuco::pair{KEY_572ef7b45c40, 1});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_572ef7b755c0(uint64_t* BUF_572ef7b8ac20, HASHTABLE_FIND HT_572ef7b45c40, HASHTABLE_PROBE HT_572ef7b8ac20, DBDecimalType* aggr0__tmp_attr0, DBI32Type* lineorder__lo_discount, DBDecimalType* lineorder__lo_extendedprice, DBI32Type* lineorder__lo_orderdate, DBI32Type* lineorder__lo_quantity, size_t lineorder_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineorder_size) return;
__shared__ DBDecimalType sh;
sh = 0;
__syncthreads();

auto reg_lineorder__lo_discount = lineorder__lo_discount[tid];
if ((evaluatePredicate(reg_lineorder__lo_discount, 1, Predicate::gte) && evaluatePredicate(reg_lineorder__lo_discount, 3, Predicate::lte))) {

    auto reg_lineorder__lo_quantity = lineorder__lo_quantity[tid];
    if ((evaluatePredicate(reg_lineorder__lo_quantity, 25, Predicate::lt))) {

        if (!(!(false))) return;
        if (!(!(false))) return;
        if (!(!(false))) return;
        if (!(!(false))) return;
        if (!(!(false))) return;
        if (!(!(false))) return;
        uint64_t KEY_572ef7b8ac20 = 0;
        auto reg_lineorder__lo_orderdate = lineorder__lo_orderdate[tid];
        
        KEY_572ef7b8ac20 |= reg_lineorder__lo_orderdate;
        //Probe Hash table
        auto SLOT_572ef7b8ac20 = HT_572ef7b8ac20.find(KEY_572ef7b8ac20);
        if (SLOT_572ef7b8ac20 != HT_572ef7b8ac20.end()) {

            if (!(true)) return;
            uint64_t KEY_572ef7b45c40 = 0;
            //Aggregate in hashtable
            auto buf_idx_572ef7b45c40 = HT_572ef7b45c40.find(KEY_572ef7b45c40)->second;
            auto reg_lineorder__lo_extendedprice = lineorder__lo_extendedprice[tid];
            auto reg_map0__tmp_attr1 = (reg_lineorder__lo_extendedprice) * ((DBDecimalType)(reg_lineorder__lo_discount));
            aggregate_sum(&sh, reg_map0__tmp_attr1);
            // aggregate_sum(&aggr0__tmp_attr0[buf_idx_572ef7b45c40], reg_map0__tmp_attr1);
        }
    }
}
__syncthreads();
if (threadIdx.x == 0) {
    aggregate_sum(&aggr0__tmp_attr0[0], sh);
}
}
__global__ void count_572ef7b9df70(uint64_t* COUNT572ef7b250c0, size_t COUNT572ef7b45c40) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT572ef7b45c40) return;
//Materialize count
atomicAdd((int*)COUNT572ef7b250c0, 1);
}
__global__ void main_572ef7b9df70(size_t COUNT572ef7b45c40, DBDecimalType* MAT572ef7b250c0aggr0__tmp_attr0, uint64_t* MAT_IDX572ef7b250c0, DBDecimalType* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT572ef7b45c40) return;
//Materialize buffers
auto mat_idx572ef7b250c0 = atomicAdd((int*)MAT_IDX572ef7b250c0, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT572ef7b250c0aggr0__tmp_attr0[mat_idx572ef7b250c0] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type* d_supplier__s_suppkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_city, DBStringType* d_supplier__s_nation, DBStringType* d_supplier__s_region, DBStringType* d_supplier__s_phone, size_t supplier_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_category, DBStringType* d_part__p_brand1, DBStringType* d_part__p_color, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, size_t part_size, DBI32Type* d_lineorder__lo_orderkey, DBI32Type* d_lineorder__lo_linenumber, DBI32Type* d_lineorder__lo_custkey, DBI32Type* d_lineorder__lo_partkey, DBI32Type* d_lineorder__lo_suppkey, DBDateType* d_lineorder__lo_orderdate, DBDateType* d_lineorder__lo_commitdate, DBStringType* d_lineorder__lo_orderpriority, DBCharType* d_lineorder__lo_shippriority, DBI32Type* d_lineorder__lo_quantity, DBDecimalType* d_lineorder__lo_extendedprice, DBDecimalType* d_lineorder__lo_ordtotalprice, DBDecimalType* d_lineorder__lo_revenue, DBDecimalType* d_lineorder__lo_supplycost, DBI32Type* d_lineorder__lo_discount, DBI32Type* d_lineorder__lo_tax, DBStringType* d_lineorder__lo_shipmode, size_t lineorder_size, DBI32Type* d_date__d_datekey, DBStringType* d_date__d_date, DBStringType* d_date__d_dayofweek, DBStringType* d_date__d_month, DBI32Type* d_date__d_year, DBI32Type* d_date__d_yearmonthnum, DBStringType* d_date__d_yearmonth, DBI32Type* d_date__d_daynuminweek, DBI32Type* d_date__d_daynuminmonth, DBI32Type* d_date__d_daynuminyear, DBI32Type* d_date__d_monthnuminyear, DBI32Type* d_date__d_weeknuminyear, DBStringType* d_date__d_sellingseason, DBI32Type* d_date__d_lastdayinweekfl, DBI32Type* d_date__d_lastdayinmonthfl, DBI32Type* d_date__d_holidayfl, DBI32Type* d_date__d_weekdayfl, size_t date_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBStringType* d_customer__c_city, DBStringType* d_customer__c_nation, DBStringType* d_customer__c_region, DBStringType* d_customer__c_phone, DBStringType* d_customer__c_mktsegment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size, DBI16Type* d_part__p_brand1_encoded, DBI16Type* d_supplier__s_nation_encoded, DBI16Type* d_customer__c_city_encoded, DBI16Type* d_supplier__s_city_encoded, DBI16Type* d_customer__c_nation_encoded, DBI16Type* d_part__p_category_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand1_map, std::unordered_map<DBI16Type, std::string>& supplier__s_nation_map, std::unordered_map<DBI16Type, std::string>& customer__c_city_map, std::unordered_map<DBI16Type, std::string>& supplier__s_city_map, std::unordered_map<DBI16Type, std::string>& customer__c_nation_map, std::unordered_map<DBI16Type, std::string>& part__p_category_map) {
//Materialize count
uint64_t* d_COUNT572ef7b8ac20;
hipMalloc(&d_COUNT572ef7b8ac20, sizeof(uint64_t));
hipMemset(d_COUNT572ef7b8ac20, 0, sizeof(uint64_t));
count_572ef7b75a90<<<std::ceil((float)date_size/128.), 128>>>(d_COUNT572ef7b8ac20, d_date__d_year, date_size);
uint64_t COUNT572ef7b8ac20;
hipMemcpy(&COUNT572ef7b8ac20, d_COUNT572ef7b8ac20, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_572ef7b8ac20;
hipMalloc(&d_BUF_IDX_572ef7b8ac20, sizeof(uint64_t));
hipMemset(d_BUF_IDX_572ef7b8ac20, 0, sizeof(uint64_t));
uint64_t* d_BUF_572ef7b8ac20;
hipMalloc(&d_BUF_572ef7b8ac20, sizeof(uint64_t) * COUNT572ef7b8ac20 * 1);
auto d_HT_572ef7b8ac20 = cuco::static_map{ (int)COUNT572ef7b8ac20*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_572ef7b75a90<<<std::ceil((float)date_size/128.), 128>>>(d_BUF_572ef7b8ac20, d_BUF_IDX_572ef7b8ac20, d_HT_572ef7b8ac20.ref(cuco::insert), d_date__d_datekey, d_date__d_year, date_size);
//Create aggregation hash table
auto d_HT_572ef7b45c40 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_572ef7b755c0<<<std::ceil((float)lineorder_size/128.), 128>>>(d_BUF_572ef7b8ac20, d_HT_572ef7b45c40.ref(cuco::insert), d_HT_572ef7b8ac20.ref(cuco::find), d_lineorder__lo_discount, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
size_t COUNT572ef7b45c40 = d_HT_572ef7b45c40.size();
thrust::device_vector<int64_t> keys_572ef7b45c40(COUNT572ef7b45c40), vals_572ef7b45c40(COUNT572ef7b45c40);
d_HT_572ef7b45c40.retrieve_all(keys_572ef7b45c40.begin(), vals_572ef7b45c40.begin());
d_HT_572ef7b45c40.clear();
int64_t* raw_keys572ef7b45c40 = thrust::raw_pointer_cast(keys_572ef7b45c40.data());
insertKeys<<<std::ceil((float)COUNT572ef7b45c40/128.), 128>>>(raw_keys572ef7b45c40, d_HT_572ef7b45c40.ref(cuco::insert), COUNT572ef7b45c40);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT572ef7b45c40);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT572ef7b45c40);
main_572ef7b755c0<<<std::ceil((float)lineorder_size/512.), 512>>>(d_BUF_572ef7b8ac20, d_HT_572ef7b45c40.ref(cuco::find), d_HT_572ef7b8ac20.ref(cuco::find), d_aggr0__tmp_attr0, d_lineorder__lo_discount, d_lineorder__lo_extendedprice, d_lineorder__lo_orderdate, d_lineorder__lo_quantity, lineorder_size);
//Materialize count
uint64_t* d_COUNT572ef7b250c0;
hipMalloc(&d_COUNT572ef7b250c0, sizeof(uint64_t));
hipMemset(d_COUNT572ef7b250c0, 0, sizeof(uint64_t));
count_572ef7b9df70<<<std::ceil((float)COUNT572ef7b45c40/128.), 128>>>(d_COUNT572ef7b250c0, COUNT572ef7b45c40);
uint64_t COUNT572ef7b250c0;
hipMemcpy(&COUNT572ef7b250c0, d_COUNT572ef7b250c0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX572ef7b250c0;
hipMalloc(&d_MAT_IDX572ef7b250c0, sizeof(uint64_t));
hipMemset(d_MAT_IDX572ef7b250c0, 0, sizeof(uint64_t));
auto MAT572ef7b250c0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT572ef7b250c0);
DBDecimalType* d_MAT572ef7b250c0aggr0__tmp_attr0;
hipMalloc(&d_MAT572ef7b250c0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT572ef7b250c0);
main_572ef7b9df70<<<std::ceil((float)COUNT572ef7b45c40/128.), 128>>>(COUNT572ef7b45c40, d_MAT572ef7b250c0aggr0__tmp_attr0, d_MAT_IDX572ef7b250c0, d_aggr0__tmp_attr0);
hipMemcpy(MAT572ef7b250c0aggr0__tmp_attr0, d_MAT572ef7b250c0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT572ef7b250c0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT572ef7b250c0; i++) { std::cout << "" << MAT572ef7b250c0aggr0__tmp_attr0[i];
std::cout << std::endl; }
hipFree(d_BUF_572ef7b8ac20);
hipFree(d_BUF_IDX_572ef7b8ac20);
hipFree(d_COUNT572ef7b8ac20);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT572ef7b250c0);
hipFree(d_MAT572ef7b250c0aggr0__tmp_attr0);
hipFree(d_MAT_IDX572ef7b250c0);
free(MAT572ef7b250c0aggr0__tmp_attr0);
}