#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_1(uint64_t* COUNT0, DBStringType* part__p_brand, DBStringType* part__p_container, DBI32Type* part__p_size, size_t part_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBI32Type reg_part__p_size[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_size[ITEM] = part__p_size[ITEM*TB + tid];
}
DBStringType reg_part__p_brand[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_brand[ITEM] = part__p_brand[ITEM*TB + tid];
}
DBStringType reg_part__p_container[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_container[ITEM] = part__p_container[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq))) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq))) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq))) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq))) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq))) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq))) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq))) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))));
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT0, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBStringType* part__p_brand, DBStringType* part__p_container, DBI32Type* part__p_partkey, DBI32Type* part__p_size, size_t part_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBI32Type reg_part__p_size[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_size[ITEM] = part__p_size[ITEM*TB + tid];
}
DBStringType reg_part__p_brand[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_brand[ITEM] = part__p_brand[ITEM*TB + tid];
}
DBStringType reg_part__p_container[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_container[ITEM] = part__p_container[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq))) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq))) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq))) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq))) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq))) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq))) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq))) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))));
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0[ITEM], buf_idx_0});
BUF_0[(buf_idx_0) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_3(uint64_t* BUF_0, HASHTABLE_PROBE HT_0, HASHTABLE_INSERT HT_2, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, DBStringType* lineitem__l_shipinstruct, DBStringType* lineitem__l_shipmode, size_t lineitem_size, DBStringType* part__p_brand, DBStringType* part__p_container, DBI32Type* part__p_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_lineitem__l_shipinstruct[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipinstruct[ITEM] = lineitem__l_shipinstruct[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_shipinstruct[ITEM], "DELIVER IN PERSON", Predicate::eq);
}
DBDecimalType reg_lineitem__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_quantity[ITEM] = lineitem__l_quantity[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 1.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 11.00, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 10.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 30.00, Predicate::lte));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 1.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 11.00, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 10.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 30.00, Predicate::lte));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 1.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 11.00, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 10.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 30.00, Predicate::lte));
}
DBStringType reg_lineitem__l_shipmode[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipmode[ITEM] = lineitem__l_shipmode[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_lineitem__l_shipmode[ITEM], "AIR", Predicate::eq)) || (evaluatePredicate(reg_lineitem__l_shipmode[ITEM], "AIR REG", Predicate::eq));
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
DBI32Type reg_part__p_size[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_size[ITEM] = part__p_size[BUF_0[slot_second0[ITEM] * 1 + 0]];
}
DBStringType reg_part__p_brand[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_brand[ITEM] = part__p_brand[BUF_0[slot_second0[ITEM] * 1 + 0]];
}
DBStringType reg_part__p_container[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_container[ITEM] = part__p_container[BUF_0[slot_second0[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 1.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 11.00, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq))) && (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 10.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::lte)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)) && (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 30.00, Predicate::lte)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))))) && (true);
}
uint64_t KEY_2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_2.insert(cuco::pair{KEY_2[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_3(uint64_t* BUF_0, HASHTABLE_PROBE HT_0, HASHTABLE_FIND HT_2, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, DBStringType* lineitem__l_shipinstruct, DBStringType* lineitem__l_shipmode, size_t lineitem_size, DBStringType* part__p_brand, DBStringType* part__p_container, DBI32Type* part__p_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_lineitem__l_shipinstruct[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipinstruct[ITEM] = lineitem__l_shipinstruct[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_shipinstruct[ITEM], "DELIVER IN PERSON", Predicate::eq);
}
DBDecimalType reg_lineitem__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_quantity[ITEM] = lineitem__l_quantity[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 1.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 11.00, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 10.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 30.00, Predicate::lte));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 1.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 11.00, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 10.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 30.00, Predicate::lte));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 1.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 11.00, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 10.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::lte)) || (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 30.00, Predicate::lte));
}
DBStringType reg_lineitem__l_shipmode[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipmode[ITEM] = lineitem__l_shipmode[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_lineitem__l_shipmode[ITEM], "AIR", Predicate::eq)) || (evaluatePredicate(reg_lineitem__l_shipmode[ITEM], "AIR REG", Predicate::eq));
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
DBI32Type reg_part__p_size[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_size[ITEM] = part__p_size[BUF_0[slot_second0[ITEM] * 1 + 0]];
}
DBStringType reg_part__p_brand[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_brand[ITEM] = part__p_brand[BUF_0[slot_second0[ITEM] * 1 + 0]];
}
DBStringType reg_part__p_container[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_container[ITEM] = part__p_container[BUF_0[slot_second0[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (((evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 5, Predicate::lte)) && (evaluatePredicate(reg_part__p_brand[ITEM], "Brand#12", Predicate::eq)) && (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 1.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 11.00, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "SM CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "SM PKG", Predicate::eq)))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq)) && ((evaluatePredicate(reg_part__p_container[ITEM], "MED BAG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PKG", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "MED PACK", Predicate::eq))) && (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 10.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::lte)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 10, Predicate::lte))) || ((evaluatePredicate(reg_part__p_brand[ITEM], "Brand#34", Predicate::eq)) && (evaluatePredicate(reg_lineitem__l_quantity[ITEM], 20.0, Predicate::gte) && evaluatePredicate(reg_lineitem__l_quantity[ITEM], 30.00, Predicate::lte)) && (evaluatePredicate(reg_part__p_size[ITEM], 1, Predicate::gte) && evaluatePredicate(reg_part__p_size[ITEM], 15, Predicate::lte)) && ((evaluatePredicate(reg_part__p_container[ITEM], "LG CASE", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG BOX", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PACK", Predicate::eq)) || (evaluatePredicate(reg_part__p_container[ITEM], "LG PKG", Predicate::eq))))) && (true);
}
uint64_t KEY_2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_2 = HT_2.find(KEY_2[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr0[buf_idx_2], reg_map0__tmp_attr1[ITEM]);
}
}
__global__ void count_5(size_t COUNT2, uint64_t* COUNT4) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT4, 1);
}
}
__global__ void main_5(size_t COUNT2, DBDecimalType* MAT4aggr0__tmp_attr0, uint64_t* MAT_IDX4, DBDecimalType* aggr0__tmp_attr0) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize buffers
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx4 = atomicAdd((int*)MAT_IDX4, 1);
MAT4aggr0__tmp_attr0[mat_idx4] = reg_aggr0__tmp_attr0[ITEM];
}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto start = std::chrono::high_resolution_clock::now();
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)part_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT0, d_part__p_brand, d_part__p_container, d_part__p_size, part_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)part_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_part__p_brand, d_part__p_container, d_part__p_partkey, d_part__p_size, part_size);
//Create aggregation hash table
auto d_HT_2 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_3<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::insert), d_lineitem__l_partkey, d_lineitem__l_quantity, d_lineitem__l_shipinstruct, d_lineitem__l_shipmode, lineitem_size, d_part__p_brand, d_part__p_container, d_part__p_size);
size_t COUNT2 = d_HT_2.size();
thrust::device_vector<int64_t> keys_2(COUNT2), vals_2(COUNT2);
d_HT_2.retrieve_all(keys_2.begin(), vals_2.begin());
d_HT_2.clear();
int64_t* raw_keys2 = thrust::raw_pointer_cast(keys_2.data());
insertKeys<<<std::ceil((float)COUNT2/128.), 128>>>(raw_keys2, d_HT_2.ref(cuco::insert), COUNT2);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT2);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT2);
main_3<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::find), d_aggr0__tmp_attr0, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_partkey, d_lineitem__l_quantity, d_lineitem__l_shipinstruct, d_lineitem__l_shipmode, lineitem_size, d_part__p_brand, d_part__p_container, d_part__p_size);
//Materialize count
uint64_t* d_COUNT4;
hipMalloc(&d_COUNT4, sizeof(uint64_t));
hipMemset(d_COUNT4, 0, sizeof(uint64_t));
count_5<<<std::ceil((float)COUNT2/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT2, d_COUNT4);
uint64_t COUNT4;
hipMemcpy(&COUNT4, d_COUNT4, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX4;
hipMalloc(&d_MAT_IDX4, sizeof(uint64_t));
hipMemset(d_MAT_IDX4, 0, sizeof(uint64_t));
auto MAT4aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT4);
DBDecimalType* d_MAT4aggr0__tmp_attr0;
hipMalloc(&d_MAT4aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT4);
main_5<<<std::ceil((float)COUNT2/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT2, d_MAT4aggr0__tmp_attr0, d_MAT_IDX4, d_aggr0__tmp_attr0);
hipMemcpy(MAT4aggr0__tmp_attr0, d_MAT4aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT4, hipMemcpyDeviceToHost);
auto end = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < COUNT4; i++) { std::cout << "" << MAT4aggr0__tmp_attr0[i];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
      size_t aux_mem = usedGpuMem() - used_mem;
      std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_COUNT0);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT4);
hipFree(d_MAT4aggr0__tmp_attr0);
hipFree(d_MAT_IDX4);
free(MAT4aggr0__tmp_attr0);
}