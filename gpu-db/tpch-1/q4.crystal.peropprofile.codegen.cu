#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_1(uint64_t* COUNT4, DBDateType* lineitem__l_commitdate, DBDateType* lineitem__l_receiptdate, size_t lineitem_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_lineitem__l_commitdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_commitdate[ITEM] = lineitem__l_commitdate[ITEM*TB + tid];
}
DBDateType reg_lineitem__l_receiptdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_receiptdate[ITEM] = lineitem__l_receiptdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_commitdate[ITEM], reg_lineitem__l_receiptdate[ITEM], Predicate::lt);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT4, 1);
}
}
template<typename HASHTABLE_INSERT_SJ>
__global__ void main_1(HASHTABLE_INSERT_SJ HT_4, int64_t* cycles_per_warp_main_1_selection_0, int64_t* cycles_per_warp_main_1_semi_join_build_4, DBDateType* lineitem__l_commitdate, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_receiptdate, size_t lineitem_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBDateType reg_lineitem__l_commitdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_commitdate[ITEM] = lineitem__l_commitdate[ITEM*TB + tid];
}
DBDateType reg_lineitem__l_receiptdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_receiptdate[ITEM] = lineitem__l_receiptdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_commitdate[ITEM], reg_lineitem__l_receiptdate[ITEM], Predicate::lt);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_4.insert(cuco::pair{KEY_4[ITEM], 1});
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_semi_join_build_4[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE_SJ, typename HASHTABLE_INSERT>
__global__ void count_3(HASHTABLE_PROBE_SJ HT_4, HASHTABLE_INSERT HT_5, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, DBI16Type* orders__o_orderpriority_encoded, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_orders__o_orderdate[ITEM], 8582, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate[ITEM], 8674, Predicate::lt);
}
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_orders__o_orderkey[ITEM];
}
//Probe Hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0;}
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_5[ITEMS_PER_THREAD];
DBI16Type reg_orders__o_orderpriority_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderpriority_encoded[ITEM] = orders__o_orderpriority_encoded[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_5[ITEM] = 0;
KEY_5[ITEM] |= reg_orders__o_orderpriority_encoded[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_5.insert(cuco::pair{KEY_5[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE_SJ, typename HASHTABLE_FIND>
__global__ void main_3(HASHTABLE_PROBE_SJ HT_4, HASHTABLE_FIND HT_5, DBI16Type* KEY_5orders__o_orderpriority_encoded, DBI64Type* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_3_aggregation_5, int64_t* cycles_per_warp_main_3_selection_2, int64_t* cycles_per_warp_main_3_semi_join_probe_4, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, DBI16Type* orders__o_orderpriority_encoded, size_t orders_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_orders__o_orderdate[ITEM], 8582, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate[ITEM], 8674, Predicate::lt);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_orders__o_orderkey[ITEM];
}
//Probe Hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0;}
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_semi_join_probe_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_5[ITEMS_PER_THREAD];
DBI16Type reg_orders__o_orderpriority_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderpriority_encoded[ITEM] = orders__o_orderpriority_encoded[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_5[ITEM] = 0;
KEY_5[ITEM] |= reg_orders__o_orderpriority_encoded[ITEM];
}
//Aggregate in hashtable
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_5 = HT_5.find(KEY_5[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5], 1);
KEY_5orders__o_orderpriority_encoded[buf_idx_5] = reg_orders__o_orderpriority_encoded[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_aggregation_5[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_7(size_t COUNT5, uint64_t* COUNT6) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT5); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT6, 1);
}
}
__global__ void main_7(size_t COUNT5, DBI64Type* MAT6aggr0__tmp_attr0, DBI16Type* MAT6orders__o_orderpriority_encoded, uint64_t* MAT_IDX6, DBI64Type* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_7_materialize_6, DBI16Type* orders__o_orderpriority_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBI16Type reg_orders__o_orderpriority_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT5); ++ITEM) {
reg_orders__o_orderpriority_encoded[ITEM] = orders__o_orderpriority_encoded[ITEM*TB + tid];
}
DBI64Type reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT5); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT5); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx6 = atomicAdd((int*)MAT_IDX6, 1);
MAT6orders__o_orderpriority_encoded[mat_idx6] = reg_orders__o_orderpriority_encoded[ITEM];
MAT6aggr0__tmp_attr0[mat_idx6] = reg_aggr0__tmp_attr0[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_materialize_6[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
//Materialize count
uint64_t* d_COUNT4;
hipMalloc(&d_COUNT4, sizeof(uint64_t));
hipMemset(d_COUNT4, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT4, d_lineitem__l_commitdate, d_lineitem__l_receiptdate, lineitem_size);
uint64_t COUNT4;
hipMemcpy(&COUNT4, d_COUNT4, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_1_semi_join_build_4;
auto main_1_semi_join_build_4_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_semi_join_build_4, sizeof(int64_t) * main_1_semi_join_build_4_cpw_size);
hipMemset(d_cycles_per_warp_main_1_semi_join_build_4, -1, sizeof(int64_t) * main_1_semi_join_build_4_cpw_size);
// Insert hash table control;
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_HT_4.ref(cuco::insert), d_cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_semi_join_build_4, d_lineitem__l_commitdate, d_lineitem__l_orderkey, d_lineitem__l_receiptdate, lineitem_size);
int64_t* d_cycles_per_warp_main_3_selection_2;
auto main_3_selection_2_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_selection_2, sizeof(int64_t) * main_3_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_3_selection_2, -1, sizeof(int64_t) * main_3_selection_2_cpw_size);
int64_t* d_cycles_per_warp_main_3_semi_join_probe_4;
auto main_3_semi_join_probe_4_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_semi_join_probe_4, sizeof(int64_t) * main_3_semi_join_probe_4_cpw_size);
hipMemset(d_cycles_per_warp_main_3_semi_join_probe_4, -1, sizeof(int64_t) * main_3_semi_join_probe_4_cpw_size);
//Create aggregation hash table
auto d_HT_5 = cuco::static_map{ (int)51270*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_3<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_HT_4.ref(cuco::find), d_HT_5.ref(cuco::insert), d_orders__o_orderdate, d_orders__o_orderkey, d_orders__o_orderpriority_encoded, orders_size);
size_t COUNT5 = d_HT_5.size();
thrust::device_vector<int64_t> keys_5(COUNT5), vals_5(COUNT5);
d_HT_5.retrieve_all(keys_5.begin(), vals_5.begin());
d_HT_5.clear();
int64_t* raw_keys5 = thrust::raw_pointer_cast(keys_5.data());
insertKeys<<<std::ceil((float)COUNT5/128.), 128>>>(raw_keys5, d_HT_5.ref(cuco::insert), COUNT5);
int64_t* d_cycles_per_warp_main_3_aggregation_5;
auto main_3_aggregation_5_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_aggregation_5, sizeof(int64_t) * main_3_aggregation_5_cpw_size);
hipMemset(d_cycles_per_warp_main_3_aggregation_5, -1, sizeof(int64_t) * main_3_aggregation_5_cpw_size);
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT5);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT5);
DBI16Type* d_KEY_5orders__o_orderpriority_encoded;
hipMalloc(&d_KEY_5orders__o_orderpriority_encoded, sizeof(DBI16Type) * COUNT5);
hipMemset(d_KEY_5orders__o_orderpriority_encoded, 0, sizeof(DBI16Type) * COUNT5);
main_3<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_HT_4.ref(cuco::find), d_HT_5.ref(cuco::find), d_KEY_5orders__o_orderpriority_encoded, d_aggr0__tmp_attr0, d_cycles_per_warp_main_3_aggregation_5, d_cycles_per_warp_main_3_selection_2, d_cycles_per_warp_main_3_semi_join_probe_4, d_orders__o_orderdate, d_orders__o_orderkey, d_orders__o_orderpriority_encoded, orders_size);
int64_t* cycles_per_warp_main_3_selection_2 = (int64_t*)malloc(sizeof(int64_t) * main_3_selection_2_cpw_size);
hipMemcpy(cycles_per_warp_main_3_selection_2, d_cycles_per_warp_main_3_selection_2, sizeof(int64_t) * main_3_selection_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_selection_2 ";
for (auto i=0ull; i < main_3_selection_2_cpw_size; i++) std::cout << cycles_per_warp_main_3_selection_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_semi_join_probe_4 = (int64_t*)malloc(sizeof(int64_t) * main_3_semi_join_probe_4_cpw_size);
hipMemcpy(cycles_per_warp_main_3_semi_join_probe_4, d_cycles_per_warp_main_3_semi_join_probe_4, sizeof(int64_t) * main_3_semi_join_probe_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_semi_join_probe_4 ";
for (auto i=0ull; i < main_3_semi_join_probe_4_cpw_size; i++) std::cout << cycles_per_warp_main_3_semi_join_probe_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_aggregation_5 = (int64_t*)malloc(sizeof(int64_t) * main_3_aggregation_5_cpw_size);
hipMemcpy(cycles_per_warp_main_3_aggregation_5, d_cycles_per_warp_main_3_aggregation_5, sizeof(int64_t) * main_3_aggregation_5_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_aggregation_5 ";
for (auto i=0ull; i < main_3_aggregation_5_cpw_size; i++) std::cout << cycles_per_warp_main_3_aggregation_5[i] << " ";
std::cout << std::endl;
//Materialize count
uint64_t* d_COUNT6;
hipMalloc(&d_COUNT6, sizeof(uint64_t));
hipMemset(d_COUNT6, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)COUNT5/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT5, d_COUNT6);
uint64_t COUNT6;
hipMemcpy(&COUNT6, d_COUNT6, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_7_materialize_6;
auto main_7_materialize_6_cpw_size = std::ceil((float)COUNT5/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_materialize_6, sizeof(int64_t) * main_7_materialize_6_cpw_size);
hipMemset(d_cycles_per_warp_main_7_materialize_6, -1, sizeof(int64_t) * main_7_materialize_6_cpw_size);
//Materialize buffers
uint64_t* d_MAT_IDX6;
hipMalloc(&d_MAT_IDX6, sizeof(uint64_t));
hipMemset(d_MAT_IDX6, 0, sizeof(uint64_t));
auto MAT6orders__o_orderpriority_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT6);
DBI16Type* d_MAT6orders__o_orderpriority_encoded;
hipMalloc(&d_MAT6orders__o_orderpriority_encoded, sizeof(DBI16Type) * COUNT6);
auto MAT6aggr0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT6);
DBI64Type* d_MAT6aggr0__tmp_attr0;
hipMalloc(&d_MAT6aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT6);
main_7<<<std::ceil((float)COUNT5/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT5, d_MAT6aggr0__tmp_attr0, d_MAT6orders__o_orderpriority_encoded, d_MAT_IDX6, d_aggr0__tmp_attr0, d_cycles_per_warp_main_7_materialize_6, d_KEY_5orders__o_orderpriority_encoded);
hipMemcpy(MAT6orders__o_orderpriority_encoded, d_MAT6orders__o_orderpriority_encoded, sizeof(DBI16Type) * COUNT6, hipMemcpyDeviceToHost);
hipMemcpy(MAT6aggr0__tmp_attr0, d_MAT6aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT6, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_7_materialize_6 = (int64_t*)malloc(sizeof(int64_t) * main_7_materialize_6_cpw_size);
hipMemcpy(cycles_per_warp_main_7_materialize_6, d_cycles_per_warp_main_7_materialize_6, sizeof(int64_t) * main_7_materialize_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_materialize_6 ";
for (auto i=0ull; i < main_7_materialize_6_cpw_size; i++) std::cout << cycles_per_warp_main_7_materialize_6[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
      size_t aux_mem = usedGpuMem() - used_mem;
      std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_COUNT4);
hipFree(d_KEY_5orders__o_orderpriority_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT6);
hipFree(d_MAT6aggr0__tmp_attr0);
hipFree(d_MAT6orders__o_orderpriority_encoded);
hipFree(d_MAT_IDX6);
free(MAT6aggr0__tmp_attr0);
free(MAT6orders__o_orderpriority_encoded);
}