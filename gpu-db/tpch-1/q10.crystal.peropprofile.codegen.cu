#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_1(uint64_t* COUNT4, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_orders__o_orderdate[ITEM], 8674, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate[ITEM], 8766, Predicate::lt);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT4, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_INSERT HT_4, int64_t* cycles_per_warp_main_1_join_build_4, int64_t* cycles_per_warp_main_1_selection_0, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_orders__o_orderdate[ITEM], 8674, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate[ITEM], 8766, Predicate::lt);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_orders__o_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_4 = atomicAdd((int*)BUF_IDX_4, 1);
HT_4.insert(cuco::pair{KEY_4[ITEM], buf_idx_4});
BUF_4[(buf_idx_4) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_build_4[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_6(uint64_t* COUNT5, size_t customer_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT5, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_6(uint64_t* BUF_5, uint64_t* BUF_IDX_5, HASHTABLE_INSERT HT_5, DBI32Type* customer__c_custkey, size_t customer_size, int64_t* cycles_per_warp_main_6_join_build_5) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_5[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_5[ITEM] = 0;
KEY_5[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_5 = atomicAdd((int*)BUF_IDX_5, 1);
HT_5.insert(cuco::pair{KEY_5[ITEM], buf_idx_5});
BUF_5[(buf_idx_5) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_6_join_build_5[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_8(uint64_t* COUNT7, size_t nation_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT7, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_8(uint64_t* BUF_7, uint64_t* BUF_IDX_7, HASHTABLE_INSERT HT_7, int64_t* cycles_per_warp_main_8_join_build_7, DBI32Type* nation__n_nationkey, size_t nation_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_7[ITEMS_PER_THREAD];
DBI32Type reg_nation__n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_nation__n_nationkey[ITEM] = nation__n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_7[ITEM] = 0;
KEY_7[ITEM] |= reg_nation__n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_7 = atomicAdd((int*)BUF_IDX_7, 1);
HT_7.insert(cuco::pair{KEY_7[ITEM], buf_idx_7});
BUF_7[(buf_idx_7) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_8_join_build_7[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_3(uint64_t* BUF_4, uint64_t* BUF_5, uint64_t* BUF_7, HASHTABLE_INSERT HT_10, HASHTABLE_PROBE HT_4, HASHTABLE_PROBE HT_5, HASHTABLE_PROBE HT_7, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, DBI32Type* lineitem__l_orderkey, DBCharType* lineitem__l_returnflag, size_t lineitem_size, DBI32Type* orders__o_custkey) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBCharType reg_lineitem__l_returnflag[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_returnflag[ITEM] = lineitem__l_returnflag[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_returnflag[ITEM], 'R', Predicate::eq);
}
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
int64_t slot_second4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0; continue;}
slot_second4[ITEM] = SLOT_4->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_5[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_custkey[ITEM] = orders__o_custkey[BUF_4[slot_second4[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_5[ITEM] = 0;
KEY_5[ITEM] |= reg_orders__o_custkey[ITEM];
}
int64_t slot_second5[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_5 = HT_5.find(KEY_5[ITEM]);
if (SLOT_5 == HT_5.end()) {selection_flags[ITEM] = 0; continue;}
slot_second5[ITEM] = SLOT_5->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_7[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[BUF_5[slot_second5[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_7[ITEM] = 0;
KEY_7[ITEM] |= reg_customer__c_nationkey[ITEM];
}
int64_t slot_second7[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_7 = HT_7.find(KEY_7[ITEM]);
if (SLOT_7 == HT_7.end()) {selection_flags[ITEM] = 0; continue;}
slot_second7[ITEM] = SLOT_7->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_custkey[ITEM] = customer__c_custkey[BUF_5[slot_second5[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_customer__c_custkey[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_10.insert(cuco::pair{KEY_10[ITEM], 1});
}
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_3(uint64_t* BUF_4, uint64_t* BUF_5, uint64_t* BUF_7, HASHTABLE_FIND HT_10, HASHTABLE_PROBE HT_4, HASHTABLE_PROBE HT_5, HASHTABLE_PROBE HT_7, DBI32Type* KEY_10customer__c_custkey, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr__c_acctbal, DBI16Type* aggr__n_name_encoded, DBDecimalType* customer__c_acctbal, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, int64_t* cycles_per_warp_main_3_aggregation_10, int64_t* cycles_per_warp_main_3_join_probe_4, int64_t* cycles_per_warp_main_3_join_probe_5, int64_t* cycles_per_warp_main_3_join_probe_7, int64_t* cycles_per_warp_main_3_map_9, int64_t* cycles_per_warp_main_3_selection_2, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBCharType* lineitem__l_returnflag, size_t lineitem_size, DBI16Type* nation__n_name_encoded, DBI32Type* orders__o_custkey) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBCharType reg_lineitem__l_returnflag[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_returnflag[ITEM] = lineitem__l_returnflag[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_returnflag[ITEM], 'R', Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
int64_t slot_second4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0; continue;}
slot_second4[ITEM] = SLOT_4->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_probe_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_5[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_custkey[ITEM] = orders__o_custkey[BUF_4[slot_second4[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_5[ITEM] = 0;
KEY_5[ITEM] |= reg_orders__o_custkey[ITEM];
}
int64_t slot_second5[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_5 = HT_5.find(KEY_5[ITEM]);
if (SLOT_5 == HT_5.end()) {selection_flags[ITEM] = 0; continue;}
slot_second5[ITEM] = SLOT_5->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_probe_5[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_7[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[BUF_5[slot_second5[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_7[ITEM] = 0;
KEY_7[ITEM] |= reg_customer__c_nationkey[ITEM];
}
int64_t slot_second7[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_7 = HT_7.find(KEY_7[ITEM]);
if (SLOT_7 == HT_7.end()) {selection_flags[ITEM] = 0; continue;}
slot_second7[ITEM] = SLOT_7->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_probe_7[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_map_9[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_custkey[ITEM] = customer__c_custkey[BUF_5[slot_second5[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_customer__c_custkey[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]));
}
DBI16Type reg_nation__n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_nation__n_name_encoded[ITEM] = nation__n_name_encoded[BUF_7[slot_second7[ITEM] * 1 + 0]];
}
DBDecimalType reg_customer__c_acctbal[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_acctbal[ITEM] = customer__c_acctbal[BUF_5[slot_second5[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_10 = HT_10.find(KEY_10[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr0[buf_idx_10], reg_map0__tmp_attr1[ITEM]);
aggregate_any(&aggr__n_name_encoded[buf_idx_10], reg_nation__n_name_encoded[ITEM]);
aggregate_any(&aggr__c_acctbal[buf_idx_10], reg_customer__c_acctbal[ITEM]);
KEY_10customer__c_custkey[buf_idx_10] = reg_customer__c_custkey[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_aggregation_10[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_12(size_t COUNT10, uint64_t* COUNT11) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT11, 1);
}
}
__global__ void main_12(size_t COUNT10, DBDecimalType* MAT11aggr0__tmp_attr0, DBDecimalType* MAT11aggr__c_acctbal, DBI16Type* MAT11aggr__n_name_encoded, DBI32Type* MAT11customer__c_custkey, uint64_t* MAT_IDX11, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr__c_acctbal, DBI16Type* aggr__n_name_encoded, DBI32Type* customer__c_custkey, int64_t* cycles_per_warp_main_12_materialize_11) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
DBDecimalType reg_aggr__c_acctbal[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_aggr__c_acctbal[ITEM] = aggr__c_acctbal[ITEM*TB + tid];
}
DBI16Type reg_aggr__n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
reg_aggr__n_name_encoded[ITEM] = aggr__n_name_encoded[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT10); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx11 = atomicAdd((int*)MAT_IDX11, 1);
MAT11customer__c_custkey[mat_idx11] = reg_customer__c_custkey[ITEM];
MAT11aggr0__tmp_attr0[mat_idx11] = reg_aggr0__tmp_attr0[ITEM];
MAT11aggr__c_acctbal[mat_idx11] = reg_aggr__c_acctbal[ITEM];
MAT11aggr__n_name_encoded[mat_idx11] = reg_aggr__n_name_encoded[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_12_materialize_11[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
//Materialize count
uint64_t* d_COUNT4;
hipMalloc(&d_COUNT4, sizeof(uint64_t));
hipMemset(d_COUNT4, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT4, d_orders__o_orderdate, orders_size);
uint64_t COUNT4;
hipMemcpy(&COUNT4, d_COUNT4, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_1_join_build_4;
auto main_1_join_build_4_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_build_4, sizeof(int64_t) * main_1_join_build_4_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_build_4, -1, sizeof(int64_t) * main_1_join_build_4_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 1);
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_BUF_IDX_4, d_HT_4.ref(cuco::insert), d_cycles_per_warp_main_1_join_build_4, d_cycles_per_warp_main_1_selection_0, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_build_4 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_build_4_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_build_4, d_cycles_per_warp_main_1_join_build_4, sizeof(int64_t) * main_1_join_build_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_build_4 ";
for (auto i=0ull; i < main_1_join_build_4_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_build_4[i] << " ";
std::cout << std::endl;
//Materialize count
uint64_t* d_COUNT5;
hipMalloc(&d_COUNT5, sizeof(uint64_t));
hipMemset(d_COUNT5, 0, sizeof(uint64_t));
count_6<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT5, customer_size);
uint64_t COUNT5;
hipMemcpy(&COUNT5, d_COUNT5, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_6_join_build_5;
auto main_6_join_build_5_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_6_join_build_5, sizeof(int64_t) * main_6_join_build_5_cpw_size);
hipMemset(d_cycles_per_warp_main_6_join_build_5, -1, sizeof(int64_t) * main_6_join_build_5_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_5;
hipMalloc(&d_BUF_IDX_5, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5, 0, sizeof(uint64_t));
uint64_t* d_BUF_5;
hipMalloc(&d_BUF_5, sizeof(uint64_t) * COUNT5 * 1);
auto d_HT_5 = cuco::static_map{ (int)COUNT5*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_6<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_5, d_BUF_IDX_5, d_HT_5.ref(cuco::insert), d_customer__c_custkey, customer_size, d_cycles_per_warp_main_6_join_build_5);
int64_t* cycles_per_warp_main_6_join_build_5 = (int64_t*)malloc(sizeof(int64_t) * main_6_join_build_5_cpw_size);
hipMemcpy(cycles_per_warp_main_6_join_build_5, d_cycles_per_warp_main_6_join_build_5, sizeof(int64_t) * main_6_join_build_5_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_6_join_build_5 ";
for (auto i=0ull; i < main_6_join_build_5_cpw_size; i++) std::cout << cycles_per_warp_main_6_join_build_5[i] << " ";
std::cout << std::endl;
//Materialize count
uint64_t* d_COUNT7;
hipMalloc(&d_COUNT7, sizeof(uint64_t));
hipMemset(d_COUNT7, 0, sizeof(uint64_t));
count_8<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT7, nation_size);
uint64_t COUNT7;
hipMemcpy(&COUNT7, d_COUNT7, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_8_join_build_7;
auto main_8_join_build_7_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_8_join_build_7, sizeof(int64_t) * main_8_join_build_7_cpw_size);
hipMemset(d_cycles_per_warp_main_8_join_build_7, -1, sizeof(int64_t) * main_8_join_build_7_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_7;
hipMalloc(&d_BUF_IDX_7, sizeof(uint64_t));
hipMemset(d_BUF_IDX_7, 0, sizeof(uint64_t));
uint64_t* d_BUF_7;
hipMalloc(&d_BUF_7, sizeof(uint64_t) * COUNT7 * 1);
auto d_HT_7 = cuco::static_map{ (int)COUNT7*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_8<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_7, d_BUF_IDX_7, d_HT_7.ref(cuco::insert), d_cycles_per_warp_main_8_join_build_7, d_nation__n_nationkey, nation_size);
int64_t* cycles_per_warp_main_8_join_build_7 = (int64_t*)malloc(sizeof(int64_t) * main_8_join_build_7_cpw_size);
hipMemcpy(cycles_per_warp_main_8_join_build_7, d_cycles_per_warp_main_8_join_build_7, sizeof(int64_t) * main_8_join_build_7_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_8_join_build_7 ";
for (auto i=0ull; i < main_8_join_build_7_cpw_size; i++) std::cout << cycles_per_warp_main_8_join_build_7[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_3_selection_2;
auto main_3_selection_2_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_selection_2, sizeof(int64_t) * main_3_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_3_selection_2, -1, sizeof(int64_t) * main_3_selection_2_cpw_size);
int64_t* d_cycles_per_warp_main_3_join_probe_4;
auto main_3_join_probe_4_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_probe_4, sizeof(int64_t) * main_3_join_probe_4_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_probe_4, -1, sizeof(int64_t) * main_3_join_probe_4_cpw_size);
int64_t* d_cycles_per_warp_main_3_join_probe_5;
auto main_3_join_probe_5_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_probe_5, sizeof(int64_t) * main_3_join_probe_5_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_probe_5, -1, sizeof(int64_t) * main_3_join_probe_5_cpw_size);
int64_t* d_cycles_per_warp_main_3_join_probe_7;
auto main_3_join_probe_7_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_probe_7, sizeof(int64_t) * main_3_join_probe_7_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_probe_7, -1, sizeof(int64_t) * main_3_join_probe_7_cpw_size);
int64_t* d_cycles_per_warp_main_3_map_9;
auto main_3_map_9_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_map_9, sizeof(int64_t) * main_3_map_9_cpw_size);
hipMemset(d_cycles_per_warp_main_3_map_9, -1, sizeof(int64_t) * main_3_map_9_cpw_size);
//Create aggregation hash table
auto d_HT_10 = cuco::static_map{ (int)45145*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_3<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_BUF_5, d_BUF_7, d_HT_10.ref(cuco::insert), d_HT_4.ref(cuco::find), d_HT_5.ref(cuco::find), d_HT_7.ref(cuco::find), d_customer__c_custkey, d_customer__c_nationkey, d_lineitem__l_orderkey, d_lineitem__l_returnflag, lineitem_size, d_orders__o_custkey);
size_t COUNT10 = d_HT_10.size();
thrust::device_vector<int64_t> keys_10(COUNT10), vals_10(COUNT10);
d_HT_10.retrieve_all(keys_10.begin(), vals_10.begin());
d_HT_10.clear();
int64_t* raw_keys10 = thrust::raw_pointer_cast(keys_10.data());
insertKeys<<<std::ceil((float)COUNT10/128.), 128>>>(raw_keys10, d_HT_10.ref(cuco::insert), COUNT10);
int64_t* d_cycles_per_warp_main_3_aggregation_10;
auto main_3_aggregation_10_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_aggregation_10, sizeof(int64_t) * main_3_aggregation_10_cpw_size);
hipMemset(d_cycles_per_warp_main_3_aggregation_10, -1, sizeof(int64_t) * main_3_aggregation_10_cpw_size);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT10);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT10);
DBI16Type* d_aggr__n_name_encoded;
hipMalloc(&d_aggr__n_name_encoded, sizeof(DBI16Type) * COUNT10);
hipMemset(d_aggr__n_name_encoded, 0, sizeof(DBI16Type) * COUNT10);
auto aggr__n_name_map = nation__n_name_map;
DBDecimalType* d_aggr__c_acctbal;
hipMalloc(&d_aggr__c_acctbal, sizeof(DBDecimalType) * COUNT10);
hipMemset(d_aggr__c_acctbal, 0, sizeof(DBDecimalType) * COUNT10);
DBI32Type* d_KEY_10customer__c_custkey;
hipMalloc(&d_KEY_10customer__c_custkey, sizeof(DBI32Type) * COUNT10);
hipMemset(d_KEY_10customer__c_custkey, 0, sizeof(DBI32Type) * COUNT10);
main_3<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_BUF_5, d_BUF_7, d_HT_10.ref(cuco::find), d_HT_4.ref(cuco::find), d_HT_5.ref(cuco::find), d_HT_7.ref(cuco::find), d_KEY_10customer__c_custkey, d_aggr0__tmp_attr0, d_aggr__c_acctbal, d_aggr__n_name_encoded, d_customer__c_acctbal, d_customer__c_custkey, d_customer__c_nationkey, d_cycles_per_warp_main_3_aggregation_10, d_cycles_per_warp_main_3_join_probe_4, d_cycles_per_warp_main_3_join_probe_5, d_cycles_per_warp_main_3_join_probe_7, d_cycles_per_warp_main_3_map_9, d_cycles_per_warp_main_3_selection_2, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_returnflag, lineitem_size, d_nation__n_name_encoded, d_orders__o_custkey);
int64_t* cycles_per_warp_main_3_selection_2 = (int64_t*)malloc(sizeof(int64_t) * main_3_selection_2_cpw_size);
hipMemcpy(cycles_per_warp_main_3_selection_2, d_cycles_per_warp_main_3_selection_2, sizeof(int64_t) * main_3_selection_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_selection_2 ";
for (auto i=0ull; i < main_3_selection_2_cpw_size; i++) std::cout << cycles_per_warp_main_3_selection_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_join_probe_4 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_probe_4_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_probe_4, d_cycles_per_warp_main_3_join_probe_4, sizeof(int64_t) * main_3_join_probe_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_probe_4 ";
for (auto i=0ull; i < main_3_join_probe_4_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_probe_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_join_probe_5 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_probe_5_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_probe_5, d_cycles_per_warp_main_3_join_probe_5, sizeof(int64_t) * main_3_join_probe_5_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_probe_5 ";
for (auto i=0ull; i < main_3_join_probe_5_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_probe_5[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_join_probe_7 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_probe_7_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_probe_7, d_cycles_per_warp_main_3_join_probe_7, sizeof(int64_t) * main_3_join_probe_7_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_probe_7 ";
for (auto i=0ull; i < main_3_join_probe_7_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_probe_7[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_map_9 = (int64_t*)malloc(sizeof(int64_t) * main_3_map_9_cpw_size);
hipMemcpy(cycles_per_warp_main_3_map_9, d_cycles_per_warp_main_3_map_9, sizeof(int64_t) * main_3_map_9_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_map_9 ";
for (auto i=0ull; i < main_3_map_9_cpw_size; i++) std::cout << cycles_per_warp_main_3_map_9[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_aggregation_10 = (int64_t*)malloc(sizeof(int64_t) * main_3_aggregation_10_cpw_size);
hipMemcpy(cycles_per_warp_main_3_aggregation_10, d_cycles_per_warp_main_3_aggregation_10, sizeof(int64_t) * main_3_aggregation_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_aggregation_10 ";
for (auto i=0ull; i < main_3_aggregation_10_cpw_size; i++) std::cout << cycles_per_warp_main_3_aggregation_10[i] << " ";
std::cout << std::endl;
//Materialize count
uint64_t* d_COUNT11;
hipMalloc(&d_COUNT11, sizeof(uint64_t));
hipMemset(d_COUNT11, 0, sizeof(uint64_t));
count_12<<<std::ceil((float)COUNT10/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT10, d_COUNT11);
uint64_t COUNT11;
hipMemcpy(&COUNT11, d_COUNT11, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_12_materialize_11;
auto main_12_materialize_11_cpw_size = std::ceil((float)COUNT10/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_12_materialize_11, sizeof(int64_t) * main_12_materialize_11_cpw_size);
hipMemset(d_cycles_per_warp_main_12_materialize_11, -1, sizeof(int64_t) * main_12_materialize_11_cpw_size);
//Materialize buffers
uint64_t* d_MAT_IDX11;
hipMalloc(&d_MAT_IDX11, sizeof(uint64_t));
hipMemset(d_MAT_IDX11, 0, sizeof(uint64_t));
auto MAT11customer__c_custkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT11);
DBI32Type* d_MAT11customer__c_custkey;
hipMalloc(&d_MAT11customer__c_custkey, sizeof(DBI32Type) * COUNT11);
auto MAT11aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT11);
DBDecimalType* d_MAT11aggr0__tmp_attr0;
hipMalloc(&d_MAT11aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT11);
auto MAT11aggr__c_acctbal = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT11);
DBDecimalType* d_MAT11aggr__c_acctbal;
hipMalloc(&d_MAT11aggr__c_acctbal, sizeof(DBDecimalType) * COUNT11);
auto MAT11aggr__n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT11);
DBI16Type* d_MAT11aggr__n_name_encoded;
hipMalloc(&d_MAT11aggr__n_name_encoded, sizeof(DBI16Type) * COUNT11);
main_12<<<std::ceil((float)COUNT10/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT10, d_MAT11aggr0__tmp_attr0, d_MAT11aggr__c_acctbal, d_MAT11aggr__n_name_encoded, d_MAT11customer__c_custkey, d_MAT_IDX11, d_aggr0__tmp_attr0, d_aggr__c_acctbal, d_aggr__n_name_encoded, d_KEY_10customer__c_custkey, d_cycles_per_warp_main_12_materialize_11);
hipMemcpy(MAT11customer__c_custkey, d_MAT11customer__c_custkey, sizeof(DBI32Type) * COUNT11, hipMemcpyDeviceToHost);
hipMemcpy(MAT11aggr0__tmp_attr0, d_MAT11aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT11, hipMemcpyDeviceToHost);
hipMemcpy(MAT11aggr__c_acctbal, d_MAT11aggr__c_acctbal, sizeof(DBDecimalType) * COUNT11, hipMemcpyDeviceToHost);
hipMemcpy(MAT11aggr__n_name_encoded, d_MAT11aggr__n_name_encoded, sizeof(DBI16Type) * COUNT11, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_12_materialize_11 = (int64_t*)malloc(sizeof(int64_t) * main_12_materialize_11_cpw_size);
hipMemcpy(cycles_per_warp_main_12_materialize_11, d_cycles_per_warp_main_12_materialize_11, sizeof(int64_t) * main_12_materialize_11_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_12_materialize_11 ";
for (auto i=0ull; i < main_12_materialize_11_cpw_size; i++) std::cout << cycles_per_warp_main_12_materialize_11[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
      size_t aux_mem = usedGpuMem() - used_mem;
      std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_COUNT4);
hipFree(d_BUF_5);
hipFree(d_BUF_IDX_5);
hipFree(d_COUNT5);
hipFree(d_BUF_7);
hipFree(d_BUF_IDX_7);
hipFree(d_COUNT7);
hipFree(d_KEY_10customer__c_custkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr__c_acctbal);
hipFree(d_aggr__n_name_encoded);
hipFree(d_COUNT11);
hipFree(d_MAT11aggr0__tmp_attr0);
hipFree(d_MAT11aggr__c_acctbal);
hipFree(d_MAT11aggr__n_name_encoded);
hipFree(d_MAT11customer__c_custkey);
hipFree(d_MAT_IDX11);
free(MAT11aggr0__tmp_attr0);
free(MAT11aggr__c_acctbal);
free(MAT11aggr__n_name_encoded);
free(MAT11customer__c_custkey);
}