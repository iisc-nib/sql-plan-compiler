#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_FIND>
__global__ void main_1(HASHTABLE_FIND HT_5, int* SLOT_COUNT_5, DBDecimalType* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_1_aggregation_5, int64_t* cycles_per_warp_main_1_map_4, int64_t* cycles_per_warp_main_1_selection_0, int64_t* cycles_per_warp_main_1_selection_2, int64_t* cycles_per_warp_main_1_selection_3, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBDecimalType* lineitem__l_quantity, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBDateType reg_lineitem__l_shipdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipdate[ITEM] = lineitem__l_shipdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 9131, Predicate::lt);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_discount[ITEM], 0.05, Predicate::gte) && evaluatePredicate(reg_lineitem__l_discount[ITEM], 0.07, Predicate::lte);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
DBDecimalType reg_lineitem__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_quantity[ITEM] = lineitem__l_quantity[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_quantity[ITEM], 24.0, Predicate::lt);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_3[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_map_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_5[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_5[ITEM] = 0;
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineitem__l_extendedprice[ITEM]) * (reg_lineitem__l_discount[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_5 = get_aggregation_slot(KEY_5[ITEM], HT_5, SLOT_COUNT_5);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5], reg_map0__tmp_attr1[ITEM]);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_aggregation_5[blockIdx.x] = cycles_per_warp;}
}
__global__ void main_7(size_t COUNT5, DBDecimalType* MAT6aggr0__tmp_attr0, uint64_t* MAT_IDX6, DBDecimalType* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_7_materialize_6) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT5); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT5); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx6 = atomicAdd((int*)MAT_IDX6, 1);
MAT6aggr0__tmp_attr0[mat_idx6] = reg_aggr0__tmp_attr0[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_materialize_6[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_2;
auto main_1_selection_2_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_2, -1, sizeof(int64_t) * main_1_selection_2_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_3;
auto main_1_selection_3_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_3, -1, sizeof(int64_t) * main_1_selection_3_cpw_size);
int64_t* d_cycles_per_warp_main_1_map_4;
auto main_1_map_4_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_map_4, sizeof(int64_t) * main_1_map_4_cpw_size);
hipMemset(d_cycles_per_warp_main_1_map_4, -1, sizeof(int64_t) * main_1_map_4_cpw_size);
int64_t* d_cycles_per_warp_main_1_aggregation_5;
auto main_1_aggregation_5_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_aggregation_5, sizeof(int64_t) * main_1_aggregation_5_cpw_size);
hipMemset(d_cycles_per_warp_main_1_aggregation_5, -1, sizeof(int64_t) * main_1_aggregation_5_cpw_size);
size_t COUNT5 = 1;
auto d_HT_5 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_5;
hipMalloc(&d_SLOT_COUNT_5, sizeof(int));
hipMemset(d_SLOT_COUNT_5, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5);
main_1<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TB>>>(d_HT_5.ref(cuco::insert_and_find), d_SLOT_COUNT_5, d_aggr0__tmp_attr0, d_cycles_per_warp_main_1_aggregation_5, d_cycles_per_warp_main_1_map_4, d_cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_3, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_quantity, d_lineitem__l_shipdate, lineitem_size);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_2 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_2 ";
for (auto i=0ull; i < main_1_selection_2_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_3 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_3, d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_3 ";
for (auto i=0ull; i < main_1_selection_3_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_3[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_map_4 = (int64_t*)malloc(sizeof(int64_t) * main_1_map_4_cpw_size);
hipMemcpy(cycles_per_warp_main_1_map_4, d_cycles_per_warp_main_1_map_4, sizeof(int64_t) * main_1_map_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_map_4 ";
for (auto i=0ull; i < main_1_map_4_cpw_size; i++) std::cout << cycles_per_warp_main_1_map_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_aggregation_5 = (int64_t*)malloc(sizeof(int64_t) * main_1_aggregation_5_cpw_size);
hipMemcpy(cycles_per_warp_main_1_aggregation_5, d_cycles_per_warp_main_1_aggregation_5, sizeof(int64_t) * main_1_aggregation_5_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_aggregation_5 ";
for (auto i=0ull; i < main_1_aggregation_5_cpw_size; i++) std::cout << cycles_per_warp_main_1_aggregation_5[i] << " ";
std::cout << std::endl;
COUNT5 = d_HT_5.size();
int64_t* d_cycles_per_warp_main_7_materialize_6;
auto main_7_materialize_6_cpw_size = std::ceil((float)COUNT5/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_materialize_6, sizeof(int64_t) * main_7_materialize_6_cpw_size);
hipMemset(d_cycles_per_warp_main_7_materialize_6, -1, sizeof(int64_t) * main_7_materialize_6_cpw_size);
size_t COUNT6 = COUNT5;
//Materialize buffers
uint64_t* d_MAT_IDX6;
hipMalloc(&d_MAT_IDX6, sizeof(uint64_t));
hipMemset(d_MAT_IDX6, 0, sizeof(uint64_t));
auto MAT6aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT6);
DBDecimalType* d_MAT6aggr0__tmp_attr0;
hipMalloc(&d_MAT6aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6);
main_7<<<std::ceil((float)COUNT5/(float)TILE_SIZE), TB>>>(COUNT5, d_MAT6aggr0__tmp_attr0, d_MAT_IDX6, d_aggr0__tmp_attr0, d_cycles_per_warp_main_7_materialize_6);
uint64_t MATCOUNT_6 = 0;
hipMemcpy(&MATCOUNT_6, d_MAT_IDX6, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT6aggr0__tmp_attr0, d_MAT6aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_7_materialize_6 = (int64_t*)malloc(sizeof(int64_t) * main_7_materialize_6_cpw_size);
hipMemcpy(cycles_per_warp_main_7_materialize_6, d_cycles_per_warp_main_7_materialize_6, sizeof(int64_t) * main_7_materialize_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_materialize_6 ";
for (auto i=0ull; i < main_7_materialize_6_cpw_size; i++) std::cout << cycles_per_warp_main_7_materialize_6[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_aggr0__tmp_attr0);
hipFree(d_MAT6aggr0__tmp_attr0);
hipFree(d_MAT_IDX6);
free(MAT6aggr0__tmp_attr0);
}