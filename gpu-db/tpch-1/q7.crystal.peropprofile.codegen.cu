#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_3(uint64_t* COUNT6, DBStringType* n1___n_name, size_t nation_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_n1___n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n1___n_name[ITEM] = n1___n_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n1___n_name[ITEM], "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name[ITEM], "GERMANY", Predicate::eq));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n1___n_name[ITEM], "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name[ITEM], "GERMANY", Predicate::eq));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n1___n_name[ITEM], "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name[ITEM], "GERMANY", Predicate::eq));
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT6, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_INSERT HT_6, int64_t* cycles_per_warp_main_3_join_build_6, int64_t* cycles_per_warp_main_3_selection_2, int64_t* cycles_per_warp_main_3_selection_4, int64_t* cycles_per_warp_main_3_selection_5, DBStringType* n1___n_name, DBI32Type* n1___n_nationkey, size_t nation_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_n1___n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n1___n_name[ITEM] = n1___n_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n1___n_name[ITEM], "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name[ITEM], "GERMANY", Predicate::eq));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n1___n_name[ITEM], "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name[ITEM], "GERMANY", Predicate::eq));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_selection_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n1___n_name[ITEM], "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name[ITEM], "GERMANY", Predicate::eq));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_selection_5[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_n1___n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n1___n_nationkey[ITEM] = n1___n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_n1___n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_6 = atomicAdd((int*)BUF_IDX_6, 1);
HT_6.insert(cuco::pair{KEY_6[ITEM], buf_idx_6});
BUF_6[(buf_idx_6) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_join_build_6[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_9(uint64_t* COUNT12, DBStringType* n2___n_name, size_t nation_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_n2___n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n2___n_name[ITEM] = n2___n_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n2___n_name[ITEM], "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name[ITEM], "FRANCE", Predicate::eq));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n2___n_name[ITEM], "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name[ITEM], "FRANCE", Predicate::eq));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n2___n_name[ITEM], "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name[ITEM], "FRANCE", Predicate::eq));
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT12, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_9(uint64_t* BUF_12, uint64_t* BUF_IDX_12, HASHTABLE_INSERT HT_12, int64_t* cycles_per_warp_main_9_join_build_12, int64_t* cycles_per_warp_main_9_selection_10, int64_t* cycles_per_warp_main_9_selection_11, int64_t* cycles_per_warp_main_9_selection_8, DBStringType* n2___n_name, DBI32Type* n2___n_nationkey, size_t nation_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_n2___n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n2___n_name[ITEM] = n2___n_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n2___n_name[ITEM], "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name[ITEM], "FRANCE", Predicate::eq));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_9_selection_8[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n2___n_name[ITEM], "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name[ITEM], "FRANCE", Predicate::eq));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_9_selection_10[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_n2___n_name[ITEM], "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name[ITEM], "FRANCE", Predicate::eq));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_9_selection_11[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_12[ITEMS_PER_THREAD];
DBI32Type reg_n2___n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n2___n_nationkey[ITEM] = n2___n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_12[ITEM] = 0;
KEY_12[ITEM] |= reg_n2___n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_12 = atomicAdd((int*)BUF_IDX_12, 1);
HT_12.insert(cuco::pair{KEY_12[ITEM], buf_idx_12});
BUF_12[(buf_idx_12) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_9_join_build_12[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE>
__global__ void count_13(uint64_t* BUF_12, uint64_t* COUNT14, HASHTABLE_PROBE HT_12, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_12[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_12[ITEM] = 0;
KEY_12[ITEM] |= reg_customer__c_nationkey[ITEM];
}
int64_t slot_second12[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_12 = HT_12.find(KEY_12[ITEM]);
if (SLOT_12 == HT_12.end()) {selection_flags[ITEM] = 0; continue;}
slot_second12[ITEM] = SLOT_12->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT14, 1);
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_13(uint64_t* BUF_12, uint64_t* BUF_14, uint64_t* BUF_IDX_14, HASHTABLE_PROBE HT_12, HASHTABLE_INSERT HT_14, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size, int64_t* cycles_per_warp_main_13_join_build_14, int64_t* cycles_per_warp_main_13_join_probe_12) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_12[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_12[ITEM] = 0;
KEY_12[ITEM] |= reg_customer__c_nationkey[ITEM];
}
int64_t slot_second12[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_12 = HT_12.find(KEY_12[ITEM]);
if (SLOT_12 == HT_12.end()) {selection_flags[ITEM] = 0; continue;}
slot_second12[ITEM] = SLOT_12->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_13_join_probe_12[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_14[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_14[ITEM] = 0;
KEY_14[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_14 = atomicAdd((int*)BUF_IDX_14, 1);
HT_14.insert(cuco::pair{KEY_14[ITEM], buf_idx_14});
BUF_14[(buf_idx_14) * 2 + 0] = BUF_12[slot_second12[ITEM] * 1 + 0];
BUF_14[(buf_idx_14) * 2 + 1] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_13_join_build_14[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE>
__global__ void count_15(uint64_t* BUF_14, uint64_t* COUNT16, HASHTABLE_PROBE HT_14, DBI32Type* orders__o_custkey, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_14[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_custkey[ITEM] = orders__o_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_14[ITEM] = 0;
KEY_14[ITEM] |= reg_orders__o_custkey[ITEM];
}
int64_t slot_second14[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_14 = HT_14.find(KEY_14[ITEM]);
if (SLOT_14 == HT_14.end()) {selection_flags[ITEM] = 0; continue;}
slot_second14[ITEM] = SLOT_14->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT16, 1);
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_15(uint64_t* BUF_14, uint64_t* BUF_16, uint64_t* BUF_IDX_16, HASHTABLE_PROBE HT_14, HASHTABLE_INSERT HT_16, int64_t* cycles_per_warp_main_15_join_build_16, int64_t* cycles_per_warp_main_15_join_probe_14, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_14[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_custkey[ITEM] = orders__o_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_14[ITEM] = 0;
KEY_14[ITEM] |= reg_orders__o_custkey[ITEM];
}
int64_t slot_second14[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_14 = HT_14.find(KEY_14[ITEM]);
if (SLOT_14 == HT_14.end()) {selection_flags[ITEM] = 0; continue;}
slot_second14[ITEM] = SLOT_14->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_15_join_probe_14[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_16[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_16[ITEM] = 0;
KEY_16[ITEM] |= reg_orders__o_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_16 = atomicAdd((int*)BUF_IDX_16, 1);
HT_16.insert(cuco::pair{KEY_16[ITEM], buf_idx_16});
BUF_16[(buf_idx_16) * 3 + 0] = ITEM*TB + tid;
BUF_16[(buf_idx_16) * 3 + 1] = BUF_14[slot_second14[ITEM] * 2 + 0];
BUF_16[(buf_idx_16) * 3 + 2] = BUF_14[slot_second14[ITEM] * 2 + 1];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_15_join_build_16[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE>
__global__ void count_7(uint64_t* BUF_6, uint64_t* COUNT17, HASHTABLE_PROBE HT_6, DBI32Type* supplier__s_nationkey, size_t supplier_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_nationkey[ITEM] = supplier__s_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_supplier__s_nationkey[ITEM];
}
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT17, 1);
}
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_7(uint64_t* BUF_17, uint64_t* BUF_6, uint64_t* BUF_IDX_17, HASHTABLE_INSERT HT_17, HASHTABLE_PROBE HT_6, int64_t* cycles_per_warp_main_7_join_build_17, int64_t* cycles_per_warp_main_7_join_probe_6, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_nationkey[ITEM] = supplier__s_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_supplier__s_nationkey[ITEM];
}
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_probe_6[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_17[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_17[ITEM] = 0;
KEY_17[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_17 = atomicAdd((int*)BUF_IDX_17, 1);
HT_17.insert(cuco::pair{KEY_17[ITEM], buf_idx_17});
BUF_17[(buf_idx_17) * 2 + 0] = ITEM*TB + tid;
BUF_17[(buf_idx_17) * 2 + 1] = BUF_6[slot_second6[ITEM] * 1 + 0];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_build_17[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_1(uint64_t* BUF_16, uint64_t* BUF_17, HASHTABLE_PROBE HT_16, HASHTABLE_PROBE HT_17, HASHTABLE_INSERT HT_20, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n1___n_name, DBI16Type* n1___n_name_encoded, DBStringType* n2___n_name, DBI16Type* n2___n_name_encoded) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_lineitem__l_shipdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipdate[ITEM] = lineitem__l_shipdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 9131, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 9861, Predicate::lte);
}
uint64_t KEY_16[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_16[ITEM] = 0;
KEY_16[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
int64_t slot_second16[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_16 = HT_16.find(KEY_16[ITEM]);
if (SLOT_16 == HT_16.end()) {selection_flags[ITEM] = 0; continue;}
slot_second16[ITEM] = SLOT_16->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_17[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_suppkey[ITEM] = lineitem__l_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_17[ITEM] = 0;
KEY_17[ITEM] |= reg_lineitem__l_suppkey[ITEM];
}
int64_t slot_second17[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_17 = HT_17.find(KEY_17[ITEM]);
if (SLOT_17 == HT_17.end()) {selection_flags[ITEM] = 0; continue;}
slot_second17[ITEM] = SLOT_17->second;
}
DBStringType reg_n1___n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_n1___n_name[ITEM] = n1___n_name[BUF_17[slot_second17[ITEM] * 2 + 1]];
}
DBStringType reg_n2___n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_n2___n_name[ITEM] = n2___n_name[BUF_16[slot_second16[ITEM] * 3 + 1]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (((evaluatePredicate(reg_n1___n_name[ITEM], "FRANCE", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name[ITEM], "GERMANY", Predicate::eq))) || ((evaluatePredicate(reg_n1___n_name[ITEM], "GERMANY", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name[ITEM], "FRANCE", Predicate::eq)))) && (true);
}
uint64_t KEY_20[ITEMS_PER_THREAD];
DBI16Type reg_n1___n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_n1___n_name_encoded[ITEM] = n1___n_name_encoded[BUF_17[slot_second17[ITEM] * 2 + 1]];
}
DBI16Type reg_n2___n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_n2___n_name_encoded[ITEM] = n2___n_name_encoded[BUF_16[slot_second16[ITEM] * 3 + 1]];
}
DBI64Type reg_map0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr0[ITEM] = ExtractFromDate("year", reg_lineitem__l_shipdate[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_20[ITEM] = 0;
KEY_20[ITEM] |= reg_n1___n_name_encoded[ITEM];
KEY_20[ITEM] <<= 16;
KEY_20[ITEM] |= reg_n2___n_name_encoded[ITEM];
KEY_20[ITEM] <<= 32;
KEY_20[ITEM] |= (DBI32Type)reg_map0__tmp_attr0[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_20.insert(cuco::pair{KEY_20[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_1(uint64_t* BUF_16, uint64_t* BUF_17, HASHTABLE_PROBE HT_16, HASHTABLE_PROBE HT_17, HASHTABLE_FIND HT_20, DBI64Type* KEY_20map0__tmp_attr0, DBI16Type* KEY_20n1___n_name_encoded, DBI16Type* KEY_20n2___n_name_encoded, DBDecimalType* aggr0__tmp_attr2, int64_t* cycles_per_warp_main_1_aggregation_20, int64_t* cycles_per_warp_main_1_join_probe_16, int64_t* cycles_per_warp_main_1_join_probe_17, int64_t* cycles_per_warp_main_1_map_18, int64_t* cycles_per_warp_main_1_map_19, int64_t* cycles_per_warp_main_1_selection_0, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n1___n_name, DBI16Type* n1___n_name_encoded, DBStringType* n2___n_name, DBI16Type* n2___n_name_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBDateType reg_lineitem__l_shipdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipdate[ITEM] = lineitem__l_shipdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 9131, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 9861, Predicate::lte);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_16[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_16[ITEM] = 0;
KEY_16[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
int64_t slot_second16[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_16 = HT_16.find(KEY_16[ITEM]);
if (SLOT_16 == HT_16.end()) {selection_flags[ITEM] = 0; continue;}
slot_second16[ITEM] = SLOT_16->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_probe_16[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_17[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_suppkey[ITEM] = lineitem__l_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_17[ITEM] = 0;
KEY_17[ITEM] |= reg_lineitem__l_suppkey[ITEM];
}
int64_t slot_second17[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_17 = HT_17.find(KEY_17[ITEM]);
if (SLOT_17 == HT_17.end()) {selection_flags[ITEM] = 0; continue;}
slot_second17[ITEM] = SLOT_17->second;
}
DBStringType reg_n1___n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_n1___n_name[ITEM] = n1___n_name[BUF_17[slot_second17[ITEM] * 2 + 1]];
}
DBStringType reg_n2___n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_n2___n_name[ITEM] = n2___n_name[BUF_16[slot_second16[ITEM] * 3 + 1]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (((evaluatePredicate(reg_n1___n_name[ITEM], "FRANCE", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name[ITEM], "GERMANY", Predicate::eq))) || ((evaluatePredicate(reg_n1___n_name[ITEM], "GERMANY", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name[ITEM], "FRANCE", Predicate::eq)))) && (true);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_probe_17[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_map_18[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_map_19[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_20[ITEMS_PER_THREAD];
DBI16Type reg_n1___n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_n1___n_name_encoded[ITEM] = n1___n_name_encoded[BUF_17[slot_second17[ITEM] * 2 + 1]];
}
DBI16Type reg_n2___n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_n2___n_name_encoded[ITEM] = n2___n_name_encoded[BUF_16[slot_second16[ITEM] * 3 + 1]];
}
DBI64Type reg_map0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr0[ITEM] = ExtractFromDate("year", reg_lineitem__l_shipdate[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_20[ITEM] = 0;
KEY_20[ITEM] |= reg_n1___n_name_encoded[ITEM];
KEY_20[ITEM] <<= 16;
KEY_20[ITEM] |= reg_n2___n_name_encoded[ITEM];
KEY_20[ITEM] <<= 32;
KEY_20[ITEM] |= (DBI32Type)reg_map0__tmp_attr0[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_20 = HT_20.find(KEY_20[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr2[buf_idx_20], reg_map0__tmp_attr1[ITEM]);
KEY_20n1___n_name_encoded[buf_idx_20] = reg_n1___n_name_encoded[ITEM];
KEY_20n2___n_name_encoded[buf_idx_20] = reg_n2___n_name_encoded[ITEM];
KEY_20map0__tmp_attr0[buf_idx_20] = reg_map0__tmp_attr0[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_aggregation_20[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_22(size_t COUNT20, uint64_t* COUNT21) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT20); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT21, 1);
}
}
__global__ void main_22(size_t COUNT20, DBDecimalType* MAT21aggr0__tmp_attr2, DBI64Type* MAT21map0__tmp_attr0, DBI16Type* MAT21n1___n_name_encoded, DBI16Type* MAT21n2___n_name_encoded, uint64_t* MAT_IDX21, DBDecimalType* aggr0__tmp_attr2, int64_t* cycles_per_warp_main_22_materialize_21, DBI64Type* map0__tmp_attr0, DBI16Type* n1___n_name_encoded, DBI16Type* n2___n_name_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBI16Type reg_n1___n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT20); ++ITEM) {
reg_n1___n_name_encoded[ITEM] = n1___n_name_encoded[ITEM*TB + tid];
}
DBI16Type reg_n2___n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT20); ++ITEM) {
reg_n2___n_name_encoded[ITEM] = n2___n_name_encoded[ITEM*TB + tid];
}
DBI64Type reg_map0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT20); ++ITEM) {
reg_map0__tmp_attr0[ITEM] = map0__tmp_attr0[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT20); ++ITEM) {
reg_aggr0__tmp_attr2[ITEM] = aggr0__tmp_attr2[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT20); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx21 = atomicAdd((int*)MAT_IDX21, 1);
MAT21n1___n_name_encoded[mat_idx21] = reg_n1___n_name_encoded[ITEM];
MAT21n2___n_name_encoded[mat_idx21] = reg_n2___n_name_encoded[ITEM];
MAT21map0__tmp_attr0[mat_idx21] = reg_map0__tmp_attr0[ITEM];
MAT21aggr0__tmp_attr2[mat_idx21] = reg_aggr0__tmp_attr2[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_22_materialize_21[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_3_selection_2;
auto main_3_selection_2_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_selection_2, sizeof(int64_t) * main_3_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_3_selection_2, -1, sizeof(int64_t) * main_3_selection_2_cpw_size);
int64_t* d_cycles_per_warp_main_3_selection_4;
auto main_3_selection_4_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_selection_4, sizeof(int64_t) * main_3_selection_4_cpw_size);
hipMemset(d_cycles_per_warp_main_3_selection_4, -1, sizeof(int64_t) * main_3_selection_4_cpw_size);
int64_t* d_cycles_per_warp_main_3_selection_5;
auto main_3_selection_5_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_selection_5, sizeof(int64_t) * main_3_selection_5_cpw_size);
hipMemset(d_cycles_per_warp_main_3_selection_5, -1, sizeof(int64_t) * main_3_selection_5_cpw_size);
//Materialize count
uint64_t* d_COUNT6;
hipMalloc(&d_COUNT6, sizeof(uint64_t));
hipMemset(d_COUNT6, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT6, d_nation__n_name, nation_size);
uint64_t COUNT6;
hipMemcpy(&COUNT6, d_COUNT6, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_3_join_build_6;
auto main_3_join_build_6_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_join_build_6, sizeof(int64_t) * main_3_join_build_6_cpw_size);
hipMemset(d_cycles_per_warp_main_3_join_build_6, -1, sizeof(int64_t) * main_3_join_build_6_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 1);
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_6, d_BUF_IDX_6, d_HT_6.ref(cuco::insert), d_cycles_per_warp_main_3_join_build_6, d_cycles_per_warp_main_3_selection_2, d_cycles_per_warp_main_3_selection_4, d_cycles_per_warp_main_3_selection_5, d_nation__n_name, d_nation__n_nationkey, nation_size);
int64_t* cycles_per_warp_main_3_selection_2 = (int64_t*)malloc(sizeof(int64_t) * main_3_selection_2_cpw_size);
hipMemcpy(cycles_per_warp_main_3_selection_2, d_cycles_per_warp_main_3_selection_2, sizeof(int64_t) * main_3_selection_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_selection_2 ";
for (auto i=0ull; i < main_3_selection_2_cpw_size; i++) std::cout << cycles_per_warp_main_3_selection_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_selection_4 = (int64_t*)malloc(sizeof(int64_t) * main_3_selection_4_cpw_size);
hipMemcpy(cycles_per_warp_main_3_selection_4, d_cycles_per_warp_main_3_selection_4, sizeof(int64_t) * main_3_selection_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_selection_4 ";
for (auto i=0ull; i < main_3_selection_4_cpw_size; i++) std::cout << cycles_per_warp_main_3_selection_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_selection_5 = (int64_t*)malloc(sizeof(int64_t) * main_3_selection_5_cpw_size);
hipMemcpy(cycles_per_warp_main_3_selection_5, d_cycles_per_warp_main_3_selection_5, sizeof(int64_t) * main_3_selection_5_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_selection_5 ";
for (auto i=0ull; i < main_3_selection_5_cpw_size; i++) std::cout << cycles_per_warp_main_3_selection_5[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_3_join_build_6 = (int64_t*)malloc(sizeof(int64_t) * main_3_join_build_6_cpw_size);
hipMemcpy(cycles_per_warp_main_3_join_build_6, d_cycles_per_warp_main_3_join_build_6, sizeof(int64_t) * main_3_join_build_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_3_join_build_6 ";
for (auto i=0ull; i < main_3_join_build_6_cpw_size; i++) std::cout << cycles_per_warp_main_3_join_build_6[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_9_selection_8;
auto main_9_selection_8_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_9_selection_8, sizeof(int64_t) * main_9_selection_8_cpw_size);
hipMemset(d_cycles_per_warp_main_9_selection_8, -1, sizeof(int64_t) * main_9_selection_8_cpw_size);
int64_t* d_cycles_per_warp_main_9_selection_10;
auto main_9_selection_10_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_9_selection_10, sizeof(int64_t) * main_9_selection_10_cpw_size);
hipMemset(d_cycles_per_warp_main_9_selection_10, -1, sizeof(int64_t) * main_9_selection_10_cpw_size);
int64_t* d_cycles_per_warp_main_9_selection_11;
auto main_9_selection_11_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_9_selection_11, sizeof(int64_t) * main_9_selection_11_cpw_size);
hipMemset(d_cycles_per_warp_main_9_selection_11, -1, sizeof(int64_t) * main_9_selection_11_cpw_size);
//Materialize count
uint64_t* d_COUNT12;
hipMalloc(&d_COUNT12, sizeof(uint64_t));
hipMemset(d_COUNT12, 0, sizeof(uint64_t));
count_9<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT12, d_nation__n_name, nation_size);
uint64_t COUNT12;
hipMemcpy(&COUNT12, d_COUNT12, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_9_join_build_12;
auto main_9_join_build_12_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_9_join_build_12, sizeof(int64_t) * main_9_join_build_12_cpw_size);
hipMemset(d_cycles_per_warp_main_9_join_build_12, -1, sizeof(int64_t) * main_9_join_build_12_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_12;
hipMalloc(&d_BUF_IDX_12, sizeof(uint64_t));
hipMemset(d_BUF_IDX_12, 0, sizeof(uint64_t));
uint64_t* d_BUF_12;
hipMalloc(&d_BUF_12, sizeof(uint64_t) * COUNT12 * 1);
auto d_HT_12 = cuco::static_map{ (int)COUNT12*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_9<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_12, d_BUF_IDX_12, d_HT_12.ref(cuco::insert), d_cycles_per_warp_main_9_join_build_12, d_cycles_per_warp_main_9_selection_10, d_cycles_per_warp_main_9_selection_11, d_cycles_per_warp_main_9_selection_8, d_nation__n_name, d_nation__n_nationkey, nation_size);
int64_t* cycles_per_warp_main_9_selection_8 = (int64_t*)malloc(sizeof(int64_t) * main_9_selection_8_cpw_size);
hipMemcpy(cycles_per_warp_main_9_selection_8, d_cycles_per_warp_main_9_selection_8, sizeof(int64_t) * main_9_selection_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_9_selection_8 ";
for (auto i=0ull; i < main_9_selection_8_cpw_size; i++) std::cout << cycles_per_warp_main_9_selection_8[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_9_selection_10 = (int64_t*)malloc(sizeof(int64_t) * main_9_selection_10_cpw_size);
hipMemcpy(cycles_per_warp_main_9_selection_10, d_cycles_per_warp_main_9_selection_10, sizeof(int64_t) * main_9_selection_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_9_selection_10 ";
for (auto i=0ull; i < main_9_selection_10_cpw_size; i++) std::cout << cycles_per_warp_main_9_selection_10[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_9_selection_11 = (int64_t*)malloc(sizeof(int64_t) * main_9_selection_11_cpw_size);
hipMemcpy(cycles_per_warp_main_9_selection_11, d_cycles_per_warp_main_9_selection_11, sizeof(int64_t) * main_9_selection_11_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_9_selection_11 ";
for (auto i=0ull; i < main_9_selection_11_cpw_size; i++) std::cout << cycles_per_warp_main_9_selection_11[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_9_join_build_12 = (int64_t*)malloc(sizeof(int64_t) * main_9_join_build_12_cpw_size);
hipMemcpy(cycles_per_warp_main_9_join_build_12, d_cycles_per_warp_main_9_join_build_12, sizeof(int64_t) * main_9_join_build_12_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_9_join_build_12 ";
for (auto i=0ull; i < main_9_join_build_12_cpw_size; i++) std::cout << cycles_per_warp_main_9_join_build_12[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_13_join_probe_12;
auto main_13_join_probe_12_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_13_join_probe_12, sizeof(int64_t) * main_13_join_probe_12_cpw_size);
hipMemset(d_cycles_per_warp_main_13_join_probe_12, -1, sizeof(int64_t) * main_13_join_probe_12_cpw_size);
//Materialize count
uint64_t* d_COUNT14;
hipMalloc(&d_COUNT14, sizeof(uint64_t));
hipMemset(d_COUNT14, 0, sizeof(uint64_t));
count_13<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_12, d_COUNT14, d_HT_12.ref(cuco::find), d_customer__c_nationkey, customer_size);
uint64_t COUNT14;
hipMemcpy(&COUNT14, d_COUNT14, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_13_join_build_14;
auto main_13_join_build_14_cpw_size = std::ceil((float)customer_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_13_join_build_14, sizeof(int64_t) * main_13_join_build_14_cpw_size);
hipMemset(d_cycles_per_warp_main_13_join_build_14, -1, sizeof(int64_t) * main_13_join_build_14_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_14;
hipMalloc(&d_BUF_IDX_14, sizeof(uint64_t));
hipMemset(d_BUF_IDX_14, 0, sizeof(uint64_t));
uint64_t* d_BUF_14;
hipMalloc(&d_BUF_14, sizeof(uint64_t) * COUNT14 * 2);
auto d_HT_14 = cuco::static_map{ (int)COUNT14*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_13<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_12, d_BUF_14, d_BUF_IDX_14, d_HT_12.ref(cuco::find), d_HT_14.ref(cuco::insert), d_customer__c_custkey, d_customer__c_nationkey, customer_size, d_cycles_per_warp_main_13_join_build_14, d_cycles_per_warp_main_13_join_probe_12);
int64_t* cycles_per_warp_main_13_join_probe_12 = (int64_t*)malloc(sizeof(int64_t) * main_13_join_probe_12_cpw_size);
hipMemcpy(cycles_per_warp_main_13_join_probe_12, d_cycles_per_warp_main_13_join_probe_12, sizeof(int64_t) * main_13_join_probe_12_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_13_join_probe_12 ";
for (auto i=0ull; i < main_13_join_probe_12_cpw_size; i++) std::cout << cycles_per_warp_main_13_join_probe_12[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_13_join_build_14 = (int64_t*)malloc(sizeof(int64_t) * main_13_join_build_14_cpw_size);
hipMemcpy(cycles_per_warp_main_13_join_build_14, d_cycles_per_warp_main_13_join_build_14, sizeof(int64_t) * main_13_join_build_14_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_13_join_build_14 ";
for (auto i=0ull; i < main_13_join_build_14_cpw_size; i++) std::cout << cycles_per_warp_main_13_join_build_14[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_15_join_probe_14;
auto main_15_join_probe_14_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_15_join_probe_14, sizeof(int64_t) * main_15_join_probe_14_cpw_size);
hipMemset(d_cycles_per_warp_main_15_join_probe_14, -1, sizeof(int64_t) * main_15_join_probe_14_cpw_size);
//Materialize count
uint64_t* d_COUNT16;
hipMalloc(&d_COUNT16, sizeof(uint64_t));
hipMemset(d_COUNT16, 0, sizeof(uint64_t));
count_15<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_14, d_COUNT16, d_HT_14.ref(cuco::find), d_orders__o_custkey, orders_size);
uint64_t COUNT16;
hipMemcpy(&COUNT16, d_COUNT16, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_15_join_build_16;
auto main_15_join_build_16_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_15_join_build_16, sizeof(int64_t) * main_15_join_build_16_cpw_size);
hipMemset(d_cycles_per_warp_main_15_join_build_16, -1, sizeof(int64_t) * main_15_join_build_16_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_16;
hipMalloc(&d_BUF_IDX_16, sizeof(uint64_t));
hipMemset(d_BUF_IDX_16, 0, sizeof(uint64_t));
uint64_t* d_BUF_16;
hipMalloc(&d_BUF_16, sizeof(uint64_t) * COUNT16 * 3);
auto d_HT_16 = cuco::static_map{ (int)COUNT16*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_15<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_14, d_BUF_16, d_BUF_IDX_16, d_HT_14.ref(cuco::find), d_HT_16.ref(cuco::insert), d_cycles_per_warp_main_15_join_build_16, d_cycles_per_warp_main_15_join_probe_14, d_orders__o_custkey, d_orders__o_orderkey, orders_size);
int64_t* cycles_per_warp_main_15_join_probe_14 = (int64_t*)malloc(sizeof(int64_t) * main_15_join_probe_14_cpw_size);
hipMemcpy(cycles_per_warp_main_15_join_probe_14, d_cycles_per_warp_main_15_join_probe_14, sizeof(int64_t) * main_15_join_probe_14_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_15_join_probe_14 ";
for (auto i=0ull; i < main_15_join_probe_14_cpw_size; i++) std::cout << cycles_per_warp_main_15_join_probe_14[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_15_join_build_16 = (int64_t*)malloc(sizeof(int64_t) * main_15_join_build_16_cpw_size);
hipMemcpy(cycles_per_warp_main_15_join_build_16, d_cycles_per_warp_main_15_join_build_16, sizeof(int64_t) * main_15_join_build_16_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_15_join_build_16 ";
for (auto i=0ull; i < main_15_join_build_16_cpw_size; i++) std::cout << cycles_per_warp_main_15_join_build_16[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_7_join_probe_6;
auto main_7_join_probe_6_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_probe_6, sizeof(int64_t) * main_7_join_probe_6_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_probe_6, -1, sizeof(int64_t) * main_7_join_probe_6_cpw_size);
//Materialize count
uint64_t* d_COUNT17;
hipMalloc(&d_COUNT17, sizeof(uint64_t));
hipMemset(d_COUNT17, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_6, d_COUNT17, d_HT_6.ref(cuco::find), d_supplier__s_nationkey, supplier_size);
uint64_t COUNT17;
hipMemcpy(&COUNT17, d_COUNT17, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_7_join_build_17;
auto main_7_join_build_17_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_build_17, sizeof(int64_t) * main_7_join_build_17_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_build_17, -1, sizeof(int64_t) * main_7_join_build_17_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_17;
hipMalloc(&d_BUF_IDX_17, sizeof(uint64_t));
hipMemset(d_BUF_IDX_17, 0, sizeof(uint64_t));
uint64_t* d_BUF_17;
hipMalloc(&d_BUF_17, sizeof(uint64_t) * COUNT17 * 2);
auto d_HT_17 = cuco::static_map{ (int)COUNT17*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_17, d_BUF_6, d_BUF_IDX_17, d_HT_17.ref(cuco::insert), d_HT_6.ref(cuco::find), d_cycles_per_warp_main_7_join_build_17, d_cycles_per_warp_main_7_join_probe_6, d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
int64_t* cycles_per_warp_main_7_join_probe_6 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_probe_6_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_probe_6, d_cycles_per_warp_main_7_join_probe_6, sizeof(int64_t) * main_7_join_probe_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_probe_6 ";
for (auto i=0ull; i < main_7_join_probe_6_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_probe_6[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_join_build_17 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_build_17_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_build_17, d_cycles_per_warp_main_7_join_build_17, sizeof(int64_t) * main_7_join_build_17_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_build_17 ";
for (auto i=0ull; i < main_7_join_build_17_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_build_17[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_join_probe_16;
auto main_1_join_probe_16_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_probe_16, sizeof(int64_t) * main_1_join_probe_16_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_probe_16, -1, sizeof(int64_t) * main_1_join_probe_16_cpw_size);
int64_t* d_cycles_per_warp_main_1_join_probe_17;
auto main_1_join_probe_17_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_probe_17, sizeof(int64_t) * main_1_join_probe_17_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_probe_17, -1, sizeof(int64_t) * main_1_join_probe_17_cpw_size);
int64_t* d_cycles_per_warp_main_1_map_18;
auto main_1_map_18_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_map_18, sizeof(int64_t) * main_1_map_18_cpw_size);
hipMemset(d_cycles_per_warp_main_1_map_18, -1, sizeof(int64_t) * main_1_map_18_cpw_size);
int64_t* d_cycles_per_warp_main_1_map_19;
auto main_1_map_19_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_map_19, sizeof(int64_t) * main_1_map_19_cpw_size);
hipMemset(d_cycles_per_warp_main_1_map_19, -1, sizeof(int64_t) * main_1_map_19_cpw_size);
//Create aggregation hash table
auto d_HT_20 = cuco::static_map{ (int)13634*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_1<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_16, d_BUF_17, d_HT_16.ref(cuco::find), d_HT_17.ref(cuco::find), d_HT_20.ref(cuco::insert), d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_nation__n_name_encoded, d_nation__n_name, d_nation__n_name_encoded);
size_t COUNT20 = d_HT_20.size();
thrust::device_vector<int64_t> keys_20(COUNT20), vals_20(COUNT20);
d_HT_20.retrieve_all(keys_20.begin(), vals_20.begin());
d_HT_20.clear();
int64_t* raw_keys20 = thrust::raw_pointer_cast(keys_20.data());
insertKeys<<<std::ceil((float)COUNT20/128.), 128>>>(raw_keys20, d_HT_20.ref(cuco::insert), COUNT20);
int64_t* d_cycles_per_warp_main_1_aggregation_20;
auto main_1_aggregation_20_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_aggregation_20, sizeof(int64_t) * main_1_aggregation_20_cpw_size);
hipMemset(d_cycles_per_warp_main_1_aggregation_20, -1, sizeof(int64_t) * main_1_aggregation_20_cpw_size);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT20);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT20);
DBI16Type* d_KEY_20n1___n_name_encoded;
hipMalloc(&d_KEY_20n1___n_name_encoded, sizeof(DBI16Type) * COUNT20);
hipMemset(d_KEY_20n1___n_name_encoded, 0, sizeof(DBI16Type) * COUNT20);
DBI16Type* d_KEY_20n2___n_name_encoded;
hipMalloc(&d_KEY_20n2___n_name_encoded, sizeof(DBI16Type) * COUNT20);
hipMemset(d_KEY_20n2___n_name_encoded, 0, sizeof(DBI16Type) * COUNT20);
DBI64Type* d_KEY_20map0__tmp_attr0;
hipMalloc(&d_KEY_20map0__tmp_attr0, sizeof(DBI64Type) * COUNT20);
hipMemset(d_KEY_20map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT20);
main_1<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_16, d_BUF_17, d_HT_16.ref(cuco::find), d_HT_17.ref(cuco::find), d_HT_20.ref(cuco::find), d_KEY_20map0__tmp_attr0, d_KEY_20n1___n_name_encoded, d_KEY_20n2___n_name_encoded, d_aggr0__tmp_attr2, d_cycles_per_warp_main_1_aggregation_20, d_cycles_per_warp_main_1_join_probe_16, d_cycles_per_warp_main_1_join_probe_17, d_cycles_per_warp_main_1_map_18, d_cycles_per_warp_main_1_map_19, d_cycles_per_warp_main_1_selection_0, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_nation__n_name_encoded, d_nation__n_name, d_nation__n_name_encoded);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_probe_16 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_probe_16_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_probe_16, d_cycles_per_warp_main_1_join_probe_16, sizeof(int64_t) * main_1_join_probe_16_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_probe_16 ";
for (auto i=0ull; i < main_1_join_probe_16_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_probe_16[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_probe_17 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_probe_17_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_probe_17, d_cycles_per_warp_main_1_join_probe_17, sizeof(int64_t) * main_1_join_probe_17_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_probe_17 ";
for (auto i=0ull; i < main_1_join_probe_17_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_probe_17[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_map_18 = (int64_t*)malloc(sizeof(int64_t) * main_1_map_18_cpw_size);
hipMemcpy(cycles_per_warp_main_1_map_18, d_cycles_per_warp_main_1_map_18, sizeof(int64_t) * main_1_map_18_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_map_18 ";
for (auto i=0ull; i < main_1_map_18_cpw_size; i++) std::cout << cycles_per_warp_main_1_map_18[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_map_19 = (int64_t*)malloc(sizeof(int64_t) * main_1_map_19_cpw_size);
hipMemcpy(cycles_per_warp_main_1_map_19, d_cycles_per_warp_main_1_map_19, sizeof(int64_t) * main_1_map_19_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_map_19 ";
for (auto i=0ull; i < main_1_map_19_cpw_size; i++) std::cout << cycles_per_warp_main_1_map_19[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_aggregation_20 = (int64_t*)malloc(sizeof(int64_t) * main_1_aggregation_20_cpw_size);
hipMemcpy(cycles_per_warp_main_1_aggregation_20, d_cycles_per_warp_main_1_aggregation_20, sizeof(int64_t) * main_1_aggregation_20_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_aggregation_20 ";
for (auto i=0ull; i < main_1_aggregation_20_cpw_size; i++) std::cout << cycles_per_warp_main_1_aggregation_20[i] << " ";
std::cout << std::endl;
//Materialize count
uint64_t* d_COUNT21;
hipMalloc(&d_COUNT21, sizeof(uint64_t));
hipMemset(d_COUNT21, 0, sizeof(uint64_t));
count_22<<<std::ceil((float)COUNT20/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT20, d_COUNT21);
uint64_t COUNT21;
hipMemcpy(&COUNT21, d_COUNT21, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_22_materialize_21;
auto main_22_materialize_21_cpw_size = std::ceil((float)COUNT20/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_22_materialize_21, sizeof(int64_t) * main_22_materialize_21_cpw_size);
hipMemset(d_cycles_per_warp_main_22_materialize_21, -1, sizeof(int64_t) * main_22_materialize_21_cpw_size);
//Materialize buffers
uint64_t* d_MAT_IDX21;
hipMalloc(&d_MAT_IDX21, sizeof(uint64_t));
hipMemset(d_MAT_IDX21, 0, sizeof(uint64_t));
auto MAT21n1___n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT21);
DBI16Type* d_MAT21n1___n_name_encoded;
hipMalloc(&d_MAT21n1___n_name_encoded, sizeof(DBI16Type) * COUNT21);
auto MAT21n2___n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT21);
DBI16Type* d_MAT21n2___n_name_encoded;
hipMalloc(&d_MAT21n2___n_name_encoded, sizeof(DBI16Type) * COUNT21);
auto MAT21map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT21);
DBI64Type* d_MAT21map0__tmp_attr0;
hipMalloc(&d_MAT21map0__tmp_attr0, sizeof(DBI64Type) * COUNT21);
auto MAT21aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT21);
DBDecimalType* d_MAT21aggr0__tmp_attr2;
hipMalloc(&d_MAT21aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT21);
main_22<<<std::ceil((float)COUNT20/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT20, d_MAT21aggr0__tmp_attr2, d_MAT21map0__tmp_attr0, d_MAT21n1___n_name_encoded, d_MAT21n2___n_name_encoded, d_MAT_IDX21, d_aggr0__tmp_attr2, d_cycles_per_warp_main_22_materialize_21, d_KEY_20map0__tmp_attr0, d_KEY_20n1___n_name_encoded, d_KEY_20n2___n_name_encoded);
hipMemcpy(MAT21n1___n_name_encoded, d_MAT21n1___n_name_encoded, sizeof(DBI16Type) * COUNT21, hipMemcpyDeviceToHost);
hipMemcpy(MAT21n2___n_name_encoded, d_MAT21n2___n_name_encoded, sizeof(DBI16Type) * COUNT21, hipMemcpyDeviceToHost);
hipMemcpy(MAT21map0__tmp_attr0, d_MAT21map0__tmp_attr0, sizeof(DBI64Type) * COUNT21, hipMemcpyDeviceToHost);
hipMemcpy(MAT21aggr0__tmp_attr2, d_MAT21aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT21, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_22_materialize_21 = (int64_t*)malloc(sizeof(int64_t) * main_22_materialize_21_cpw_size);
hipMemcpy(cycles_per_warp_main_22_materialize_21, d_cycles_per_warp_main_22_materialize_21, sizeof(int64_t) * main_22_materialize_21_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_22_materialize_21 ";
for (auto i=0ull; i < main_22_materialize_21_cpw_size; i++) std::cout << cycles_per_warp_main_22_materialize_21[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
      size_t aux_mem = usedGpuMem() - used_mem;
      std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_COUNT6);
hipFree(d_BUF_12);
hipFree(d_BUF_IDX_12);
hipFree(d_COUNT12);
hipFree(d_BUF_14);
hipFree(d_BUF_IDX_14);
hipFree(d_COUNT14);
hipFree(d_BUF_16);
hipFree(d_BUF_IDX_16);
hipFree(d_COUNT16);
hipFree(d_BUF_17);
hipFree(d_BUF_IDX_17);
hipFree(d_COUNT17);
hipFree(d_KEY_20map0__tmp_attr0);
hipFree(d_KEY_20n1___n_name_encoded);
hipFree(d_KEY_20n2___n_name_encoded);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_COUNT21);
hipFree(d_MAT21aggr0__tmp_attr2);
hipFree(d_MAT21map0__tmp_attr0);
hipFree(d_MAT21n1___n_name_encoded);
hipFree(d_MAT21n2___n_name_encoded);
hipFree(d_MAT_IDX21);
free(MAT21aggr0__tmp_attr2);
free(MAT21map0__tmp_attr0);
free(MAT21n1___n_name_encoded);
free(MAT21n2___n_name_encoded);
}