#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
__global__ void count_1(uint64_t* COUNT0, DBStringType* part__p_name, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_name = part__p_name[tid];
if (!(Like(reg_part__p_name, "forest", "", nullptr, nullptr, 0))) return;
//Materialize count
atomicAdd((int*)COUNT0, 1);
}
template<typename HASHTABLE_INSERT_SJ>
__global__ void main_1(HASHTABLE_INSERT_SJ HT_0, DBStringType* part__p_name, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
auto reg_part__p_name = part__p_name[tid];
if (!(Like(reg_part__p_name, "forest", "", nullptr, nullptr, 0))) return;
uint64_t KEY_0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_0 |= reg_part__p_partkey;
// Insert hash table kernel;
HT_0.insert(cuco::pair{KEY_0, 1});
}
template<typename HASHTABLE_PROBE_SJ>
__global__ void count_3(uint64_t* COUNT2, HASHTABLE_PROBE_SJ HT_0, DBI32Type* partsupp__ps_partkey, size_t partsupp_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= partsupp_size) return;
uint64_t KEY_0 = 0;
auto reg_partsupp__ps_partkey = partsupp__ps_partkey[tid];

KEY_0 |= reg_partsupp__ps_partkey;
//Probe Hash table
auto SLOT_0 = HT_0.find(KEY_0);
if (SLOT_0 == HT_0.end()) return;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT2, 1);
}
template<typename HASHTABLE_PROBE_SJ, typename HASHTABLE_INSERT_PK>
__global__ void main_3(uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_PROBE_SJ HT_0, HASHTABLE_INSERT_PK HT_2, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= partsupp_size) return;
uint64_t KEY_0 = 0;
auto reg_partsupp__ps_partkey = partsupp__ps_partkey[tid];

KEY_0 |= reg_partsupp__ps_partkey;
//Probe Hash table
auto SLOT_0 = HT_0.find(KEY_0);
if (SLOT_0 == HT_0.end()) return;
if (!(true)) return;
uint64_t KEY_2 = 0;
auto reg_partsupp__ps_suppkey = partsupp__ps_suppkey[tid];

KEY_2 |= reg_partsupp__ps_suppkey;
KEY_2 <<= 32;
KEY_2 |= reg_partsupp__ps_partkey;
// Insert hash table kernel;
auto buf_idx_2 = atomicAdd((int*)BUF_IDX_2, 1);
HT_2.insert(cuco::pair{KEY_2, buf_idx_2});
BUF_2[buf_idx_2 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_INSERT>
__global__ void count_5(uint64_t* BUF_2, HASHTABLE_PROBE_PK HT_2, HASHTABLE_INSERT HT_4, DBI32Type* lineitem__l_partkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::lt))) return;
uint64_t KEY_2 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_2 |= reg_lineitem__l_suppkey;
auto reg_lineitem__l_partkey = lineitem__l_partkey[tid];
KEY_2 <<= 32;
KEY_2 |= reg_lineitem__l_partkey;
//Probe Hash table
auto SLOT_2 = HT_2.find(KEY_2);
if (SLOT_2 == HT_2.end()) return;
if (!(true)) return;
uint64_t KEY_4 = 0;

KEY_4 |= reg_lineitem__l_suppkey;
KEY_4 <<= 32;
KEY_4 |= reg_lineitem__l_partkey;
//Create aggregation hash table
HT_4.insert(cuco::pair{KEY_4, 1});
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_FIND>
__global__ void main_5(uint64_t* BUF_2, HASHTABLE_PROBE_PK HT_2, HASHTABLE_FIND HT_4, DBI32Type* KEY_4lineitem__l_partkey, DBI32Type* KEY_4lineitem__l_suppkey, DBDecimalType* aggr0__tmp_attr0, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI32Type* moved_aggr__ps_suppkey, DBI32Type* moved_aggr_u_1__ps_partkey, DBI32Type* moved_aggr_u_2__ps_availqty, DBI32Type* partsupp__ps_availqty, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::lt))) return;
uint64_t KEY_2 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_2 |= reg_lineitem__l_suppkey;
auto reg_lineitem__l_partkey = lineitem__l_partkey[tid];
KEY_2 <<= 32;
KEY_2 |= reg_lineitem__l_partkey;
//Probe Hash table
auto SLOT_2 = HT_2.find(KEY_2);
if (SLOT_2 == HT_2.end()) return;
if (!(true)) return;
uint64_t KEY_4 = 0;

KEY_4 |= reg_lineitem__l_suppkey;
KEY_4 <<= 32;
KEY_4 |= reg_lineitem__l_partkey;
//Aggregate in hashtable
auto buf_idx_4 = HT_4.find(KEY_4)->second;
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_4], reg_lineitem__l_quantity);
auto reg_partsupp__ps_suppkey = partsupp__ps_suppkey[BUF_2[SLOT_2->second * 1 + 0]];
aggregate_any(&moved_aggr__ps_suppkey[buf_idx_4], reg_partsupp__ps_suppkey);
auto reg_partsupp__ps_partkey = partsupp__ps_partkey[BUF_2[SLOT_2->second * 1 + 0]];
aggregate_any(&moved_aggr_u_1__ps_partkey[buf_idx_4], reg_partsupp__ps_partkey);
auto reg_partsupp__ps_availqty = partsupp__ps_availqty[BUF_2[SLOT_2->second * 1 + 0]];
aggregate_any(&moved_aggr_u_2__ps_availqty[buf_idx_4], reg_partsupp__ps_availqty);
KEY_4lineitem__l_suppkey[buf_idx_4] = reg_lineitem__l_suppkey;
KEY_4lineitem__l_partkey[buf_idx_4] = reg_lineitem__l_partkey;
}
__global__ void count_7(uint64_t* COUNT6, DBStringType* nation__n_name, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_nation__n_name = nation__n_name[tid];
if (!(evaluatePredicate(reg_nation__n_name, "CANADA", Predicate::eq))) return;
//Materialize count
atomicAdd((int*)COUNT6, 1);
}
template<typename HASHTABLE_INSERT_PK>
__global__ void main_7(uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_INSERT_PK HT_6, DBStringType* nation__n_name, DBI32Type* nation__n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_nation__n_name = nation__n_name[tid];
if (!(evaluatePredicate(reg_nation__n_name, "CANADA", Predicate::eq))) return;
uint64_t KEY_6 = 0;
auto reg_nation__n_nationkey = nation__n_nationkey[tid];

KEY_6 |= reg_nation__n_nationkey;
// Insert hash table kernel;
auto buf_idx_6 = atomicAdd((int*)BUF_IDX_6, 1);
HT_6.insert(cuco::pair{KEY_6, buf_idx_6});
BUF_6[buf_idx_6 * 1 + 0] = tid;
}
__global__ void count_9(size_t COUNT4, uint64_t* COUNT8, DBDecimalType* aggr0__tmp_attr0, DBI32Type* moved_aggr__ps_suppkey, DBI32Type* moved_aggr_u_1__ps_partkey, DBI32Type* moved_aggr_u_2__ps_availqty) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT4) return;
if (!(!(false))) return;
auto reg_partsupp__ps_availqty = moved_aggr_u_2__ps_availqty[tid];
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
auto reg_map0__tmp_attr1 = (0.5) * (reg_aggr0__tmp_attr0);
if (!(((true) && (evaluatePredicate(((DBDecimalType)reg_partsupp__ps_availqty), reg_map0__tmp_attr1, Predicate::gt))) && (true))) return;
//Materialize count
atomicAdd((int*)COUNT8, 1);
}
template<typename HASHTABLE_INSERT_SJ>
__global__ void main_9(size_t COUNT4, HASHTABLE_INSERT_SJ HT_8, DBDecimalType* aggr0__tmp_attr0, DBI32Type* moved_aggr__ps_suppkey, DBI32Type* moved_aggr_u_1__ps_partkey, DBI32Type* moved_aggr_u_2__ps_availqty) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT4) return;
if (!(!(false))) return;
auto reg_partsupp__ps_availqty = moved_aggr_u_2__ps_availqty[tid];
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
auto reg_map0__tmp_attr1 = (0.5) * (reg_aggr0__tmp_attr0);
if (!(((true) && (evaluatePredicate(((DBDecimalType)reg_partsupp__ps_availqty), reg_map0__tmp_attr1, Predicate::gt))) && (true))) return;
uint64_t KEY_8 = 0;
auto reg_partsupp__ps_suppkey = moved_aggr__ps_suppkey[tid];

KEY_8 |= reg_partsupp__ps_suppkey;
// Insert hash table kernel;
HT_8.insert(cuco::pair{KEY_8, 1});
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_PROBE_SJ>
__global__ void count_11(uint64_t* BUF_6, uint64_t* COUNT10, HASHTABLE_PROBE_PK HT_6, HASHTABLE_PROBE_SJ HT_8, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_6 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_6 |= reg_supplier__s_nationkey;
//Probe Hash table
auto SLOT_6 = HT_6.find(KEY_6);
if (SLOT_6 == HT_6.end()) return;
if (!(true)) return;
uint64_t KEY_8 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_8 |= reg_supplier__s_suppkey;
//Probe Hash table
auto SLOT_8 = HT_8.find(KEY_8);
if (SLOT_8 == HT_8.end()) return;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT10, 1);
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_PROBE_SJ>
__global__ void main_11(uint64_t* BUF_6, HASHTABLE_PROBE_PK HT_6, HASHTABLE_PROBE_SJ HT_8, DBI16Type* MAT10supplier__s_address_encoded, DBI16Type* MAT10supplier__s_name_encoded, uint64_t* MAT_IDX10, DBI16Type* supplier__s_address_encoded, DBI16Type* supplier__s_name_encoded, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_6 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_6 |= reg_supplier__s_nationkey;
//Probe Hash table
auto SLOT_6 = HT_6.find(KEY_6);
if (SLOT_6 == HT_6.end()) return;
if (!(true)) return;
uint64_t KEY_8 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_8 |= reg_supplier__s_suppkey;
//Probe Hash table
auto SLOT_8 = HT_8.find(KEY_8);
if (SLOT_8 == HT_8.end()) return;
if (!(true)) return;
//Materialize buffers
auto mat_idx10 = atomicAdd((int*)MAT_IDX10, 1);
auto reg_supplier__s_name_encoded = supplier__s_name_encoded[tid];
MAT10supplier__s_name_encoded[mat_idx10] = reg_supplier__s_name_encoded;
auto reg_supplier__s_address_encoded = supplier__s_address_encoded[tid];
MAT10supplier__s_address_encoded[mat_idx10] = reg_supplier__s_address_encoded;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto start = std::chrono::high_resolution_clock::now();
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)part_size/128.), 128>>>(d_COUNT0, d_part__p_name, part_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)part_size/128.), 128>>>(d_HT_0.ref(cuco::insert), d_part__p_name, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT2;
hipMalloc(&d_COUNT2, sizeof(uint64_t));
hipMemset(d_COUNT2, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)partsupp_size/128.), 128>>>(d_COUNT2, d_HT_0.ref(cuco::find), d_partsupp__ps_partkey, partsupp_size);
uint64_t COUNT2;
hipMemcpy(&COUNT2, d_COUNT2, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 1);
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)partsupp_size/128.), 128>>>(d_BUF_2, d_BUF_IDX_2, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::insert), d_partsupp__ps_partkey, d_partsupp__ps_suppkey, partsupp_size);
//Create aggregation hash table
auto d_HT_4 = cuco::static_map{ (int)861503*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5<<<std::ceil((float)lineitem_size/128.), 128>>>(d_BUF_2, d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::insert), d_lineitem__l_partkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size);
size_t COUNT4 = d_HT_4.size();
thrust::device_vector<int64_t> keys_4(COUNT4), vals_4(COUNT4);
d_HT_4.retrieve_all(keys_4.begin(), vals_4.begin());
d_HT_4.clear();
int64_t* raw_keys4 = thrust::raw_pointer_cast(keys_4.data());
insertKeys<<<std::ceil((float)COUNT4/128.), 128>>>(raw_keys4, d_HT_4.ref(cuco::insert), COUNT4);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT4);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT4);
DBI32Type* d_moved_aggr__ps_suppkey;
hipMalloc(&d_moved_aggr__ps_suppkey, sizeof(DBI32Type) * COUNT4);
hipMemset(d_moved_aggr__ps_suppkey, 0, sizeof(DBI32Type) * COUNT4);
DBI32Type* d_moved_aggr_u_1__ps_partkey;
hipMalloc(&d_moved_aggr_u_1__ps_partkey, sizeof(DBI32Type) * COUNT4);
hipMemset(d_moved_aggr_u_1__ps_partkey, 0, sizeof(DBI32Type) * COUNT4);
DBI32Type* d_moved_aggr_u_2__ps_availqty;
hipMalloc(&d_moved_aggr_u_2__ps_availqty, sizeof(DBI32Type) * COUNT4);
hipMemset(d_moved_aggr_u_2__ps_availqty, 0, sizeof(DBI32Type) * COUNT4);
DBI32Type* d_KEY_4lineitem__l_suppkey;
hipMalloc(&d_KEY_4lineitem__l_suppkey, sizeof(DBI32Type) * COUNT4);
hipMemset(d_KEY_4lineitem__l_suppkey, 0, sizeof(DBI32Type) * COUNT4);
DBI32Type* d_KEY_4lineitem__l_partkey;
hipMalloc(&d_KEY_4lineitem__l_partkey, sizeof(DBI32Type) * COUNT4);
hipMemset(d_KEY_4lineitem__l_partkey, 0, sizeof(DBI32Type) * COUNT4);
main_5<<<std::ceil((float)lineitem_size/128.), 128>>>(d_BUF_2, d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::find), d_KEY_4lineitem__l_partkey, d_KEY_4lineitem__l_suppkey, d_aggr0__tmp_attr0, d_lineitem__l_partkey, d_lineitem__l_quantity, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_moved_aggr__ps_suppkey, d_moved_aggr_u_1__ps_partkey, d_moved_aggr_u_2__ps_availqty, d_partsupp__ps_availqty, d_partsupp__ps_partkey, d_partsupp__ps_suppkey);
//Materialize count
uint64_t* d_COUNT6;
hipMalloc(&d_COUNT6, sizeof(uint64_t));
hipMemset(d_COUNT6, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)nation_size/128.), 128>>>(d_COUNT6, d_nation__n_name, nation_size);
uint64_t COUNT6;
hipMemcpy(&COUNT6, d_COUNT6, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 1);
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)nation_size/128.), 128>>>(d_BUF_6, d_BUF_IDX_6, d_HT_6.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
//Materialize count
uint64_t* d_COUNT8;
hipMalloc(&d_COUNT8, sizeof(uint64_t));
hipMemset(d_COUNT8, 0, sizeof(uint64_t));
count_9<<<std::ceil((float)COUNT4/128.), 128>>>(COUNT4, d_COUNT8, d_aggr0__tmp_attr0, d_moved_aggr__ps_suppkey, d_moved_aggr_u_1__ps_partkey, d_moved_aggr_u_2__ps_availqty);
uint64_t COUNT8;
hipMemcpy(&COUNT8, d_COUNT8, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
auto d_HT_8 = cuco::static_map{ (int)COUNT8*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_9<<<std::ceil((float)COUNT4/128.), 128>>>(COUNT4, d_HT_8.ref(cuco::insert), d_aggr0__tmp_attr0, d_moved_aggr__ps_suppkey, d_moved_aggr_u_1__ps_partkey, d_moved_aggr_u_2__ps_availqty);
//Materialize count
uint64_t* d_COUNT10;
hipMalloc(&d_COUNT10, sizeof(uint64_t));
hipMemset(d_COUNT10, 0, sizeof(uint64_t));
count_11<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_6, d_COUNT10, d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::find), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
uint64_t COUNT10;
hipMemcpy(&COUNT10, d_COUNT10, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX10;
hipMalloc(&d_MAT_IDX10, sizeof(uint64_t));
hipMemset(d_MAT_IDX10, 0, sizeof(uint64_t));
auto MAT10supplier__s_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT10);
DBI16Type* d_MAT10supplier__s_name_encoded;
hipMalloc(&d_MAT10supplier__s_name_encoded, sizeof(DBI16Type) * COUNT10);
auto MAT10supplier__s_address_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT10);
DBI16Type* d_MAT10supplier__s_address_encoded;
hipMalloc(&d_MAT10supplier__s_address_encoded, sizeof(DBI16Type) * COUNT10);
main_11<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_6, d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::find), d_MAT10supplier__s_address_encoded, d_MAT10supplier__s_name_encoded, d_MAT_IDX10, d_supplier__s_address_encoded, d_supplier__s_name_encoded, d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
hipMemcpy(MAT10supplier__s_name_encoded, d_MAT10supplier__s_name_encoded, sizeof(DBI16Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10supplier__s_address_encoded, d_MAT10supplier__s_address_encoded, sizeof(DBI16Type) * COUNT10, hipMemcpyDeviceToHost);
auto end = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < COUNT10; i++) { std::cout << "" << supplier__s_name_map[MAT10supplier__s_name_encoded[i]];
std::cout << "|" << supplier__s_address_map[MAT10supplier__s_address_encoded[i]];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_COUNT0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_COUNT2);
hipFree(d_KEY_4lineitem__l_partkey);
hipFree(d_KEY_4lineitem__l_suppkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_moved_aggr__ps_suppkey);
hipFree(d_moved_aggr_u_1__ps_partkey);
hipFree(d_moved_aggr_u_2__ps_availqty);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_COUNT6);
hipFree(d_COUNT8);
hipFree(d_COUNT10);
hipFree(d_MAT10supplier__s_address_encoded);
hipFree(d_MAT10supplier__s_name_encoded);
hipFree(d_MAT_IDX10);
free(MAT10supplier__s_address_encoded);
free(MAT10supplier__s_name_encoded);
}