#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_FIND>
__global__ void main_1(HASHTABLE_FIND HT_4, DBCharType* KEY_4lineitem__l_linestatus, DBCharType* KEY_4lineitem__l_returnflag, int* SLOT_COUNT_4, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr0__tmp_attr1, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, DBI64Type* aggr0__tmp_attr9, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, DBDecimalType* aggr_rw__rw2, DBI64Type* aggr_rw__rw3, DBDecimalType* aggr_rw__rw4, DBI64Type* aggr_rw__rw5, int64_t* cycles_per_warp_main_1_aggregation_4, int64_t* cycles_per_warp_main_1_map_2, int64_t* cycles_per_warp_main_1_map_3, int64_t* cycles_per_warp_main_1_selection_0, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBCharType* lineitem__l_linestatus, DBDecimalType* lineitem__l_quantity, DBCharType* lineitem__l_returnflag, DBDateType* lineitem__l_shipdate, DBDecimalType* lineitem__l_tax, size_t lineitem_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBDateType reg_lineitem__l_shipdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipdate[ITEM] = lineitem__l_shipdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 10471, Predicate::lte);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_map_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_map_3[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_4[ITEMS_PER_THREAD];
DBCharType reg_lineitem__l_returnflag[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_returnflag[ITEM] = lineitem__l_returnflag[ITEM*TB + tid];
}
DBCharType reg_lineitem__l_linestatus[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_linestatus[ITEM] = lineitem__l_linestatus[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_lineitem__l_returnflag[ITEM];
KEY_4[ITEM] <<= 8;
KEY_4[ITEM] |= reg_lineitem__l_linestatus[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_tax[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_tax[ITEM] = lineitem__l_tax[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr5[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr5[ITEM] = ((reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]))) * ((1.0) + (reg_lineitem__l_tax[ITEM]));
}
DBDecimalType reg_map0__tmp_attr3[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr3[ITEM] = (reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]));
}
DBDecimalType reg_lineitem__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_quantity[ITEM] = lineitem__l_quantity[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_4 = get_aggregation_slot(KEY_4[ITEM], HT_4, SLOT_COUNT_4);
aggregate_sum(&aggr0__tmp_attr9[buf_idx_4], 1);
aggregate_sum(&aggr0__tmp_attr4[buf_idx_4], reg_map0__tmp_attr5[ITEM]);
aggregate_sum(&aggr0__tmp_attr2[buf_idx_4], reg_map0__tmp_attr3[ITEM]);
aggregate_sum(&aggr0__tmp_attr1[buf_idx_4], reg_lineitem__l_extendedprice[ITEM]);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_4], reg_lineitem__l_quantity[ITEM]);
aggregate_sum(&aggr_rw__rw0[buf_idx_4], reg_lineitem__l_discount[ITEM]);
aggregate_sum(&aggr_rw__rw1[buf_idx_4], 1);
aggregate_sum(&aggr_rw__rw2[buf_idx_4], reg_lineitem__l_extendedprice[ITEM]);
aggregate_sum(&aggr_rw__rw3[buf_idx_4], 1);
aggregate_sum(&aggr_rw__rw4[buf_idx_4], reg_lineitem__l_quantity[ITEM]);
aggregate_sum(&aggr_rw__rw5[buf_idx_4], 1);
KEY_4lineitem__l_returnflag[buf_idx_4] = reg_lineitem__l_returnflag[ITEM];
KEY_4lineitem__l_linestatus[buf_idx_4] = reg_lineitem__l_linestatus[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_aggregation_4[blockIdx.x] = cycles_per_warp;}
}
__global__ void main_6(size_t COUNT4, DBDecimalType* MAT9aggr0__tmp_attr0, DBDecimalType* MAT9aggr0__tmp_attr1, DBDecimalType* MAT9aggr0__tmp_attr2, DBDecimalType* MAT9aggr0__tmp_attr4, DBDecimalType* MAT9aggr0__tmp_attr6, DBDecimalType* MAT9aggr0__tmp_attr7, DBDecimalType* MAT9aggr0__tmp_attr8, DBI64Type* MAT9aggr0__tmp_attr9, DBCharType* MAT9lineitem__l_linestatus, DBCharType* MAT9lineitem__l_returnflag, uint64_t* MAT_IDX9, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr0__tmp_attr1, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, DBI64Type* aggr0__tmp_attr9, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, DBDecimalType* aggr_rw__rw2, DBI64Type* aggr_rw__rw3, DBDecimalType* aggr_rw__rw4, DBI64Type* aggr_rw__rw5, int64_t* cycles_per_warp_main_6_map_5, int64_t* cycles_per_warp_main_6_map_7, int64_t* cycles_per_warp_main_6_map_8, int64_t* cycles_per_warp_main_6_materialize_9, DBCharType* lineitem__l_linestatus, DBCharType* lineitem__l_returnflag) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_6_map_5[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_6_map_7[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_6_map_8[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBCharType reg_lineitem__l_returnflag[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_lineitem__l_returnflag[ITEM] = lineitem__l_returnflag[ITEM*TB + tid];
}
DBCharType reg_lineitem__l_linestatus[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_lineitem__l_linestatus[ITEM] = lineitem__l_linestatus[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr0__tmp_attr1[ITEM] = aggr0__tmp_attr1[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr0__tmp_attr2[ITEM] = aggr0__tmp_attr2[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr0__tmp_attr4[ITEM] = aggr0__tmp_attr4[ITEM*TB + tid];
}
DBI64Type reg_aggr_rw__rw5[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr_rw__rw5[ITEM] = aggr_rw__rw5[ITEM*TB + tid];
}
DBDecimalType reg_aggr_rw__rw4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr_rw__rw4[ITEM] = aggr_rw__rw4[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr0__tmp_attr6[ITEM] = (reg_aggr_rw__rw4[ITEM]) / ((DBDecimalType)(reg_aggr_rw__rw5[ITEM]));
}
DBI64Type reg_aggr_rw__rw3[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr_rw__rw3[ITEM] = aggr_rw__rw3[ITEM*TB + tid];
}
DBDecimalType reg_aggr_rw__rw2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr_rw__rw2[ITEM] = aggr_rw__rw2[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr7[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr0__tmp_attr7[ITEM] = (reg_aggr_rw__rw2[ITEM]) / ((DBDecimalType)(reg_aggr_rw__rw3[ITEM]));
}
DBI64Type reg_aggr_rw__rw1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr_rw__rw1[ITEM] = aggr_rw__rw1[ITEM*TB + tid];
}
DBDecimalType reg_aggr_rw__rw0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr_rw__rw0[ITEM] = aggr_rw__rw0[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr8[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr0__tmp_attr8[ITEM] = (reg_aggr_rw__rw0[ITEM]) / ((DBDecimalType)(reg_aggr_rw__rw1[ITEM]));
}
DBI64Type reg_aggr0__tmp_attr9[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr0__tmp_attr9[ITEM] = aggr0__tmp_attr9[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx9 = atomicAdd((int*)MAT_IDX9, 1);
MAT9lineitem__l_returnflag[mat_idx9] = reg_lineitem__l_returnflag[ITEM];
MAT9lineitem__l_linestatus[mat_idx9] = reg_lineitem__l_linestatus[ITEM];
MAT9aggr0__tmp_attr0[mat_idx9] = reg_aggr0__tmp_attr0[ITEM];
MAT9aggr0__tmp_attr1[mat_idx9] = reg_aggr0__tmp_attr1[ITEM];
MAT9aggr0__tmp_attr2[mat_idx9] = reg_aggr0__tmp_attr2[ITEM];
MAT9aggr0__tmp_attr4[mat_idx9] = reg_aggr0__tmp_attr4[ITEM];
MAT9aggr0__tmp_attr6[mat_idx9] = reg_aggr0__tmp_attr6[ITEM];
MAT9aggr0__tmp_attr7[mat_idx9] = reg_aggr0__tmp_attr7[ITEM];
MAT9aggr0__tmp_attr8[mat_idx9] = reg_aggr0__tmp_attr8[ITEM];
MAT9aggr0__tmp_attr9[mat_idx9] = reg_aggr0__tmp_attr9[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_6_materialize_9[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_map_2;
auto main_1_map_2_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_map_2, sizeof(int64_t) * main_1_map_2_cpw_size);
hipMemset(d_cycles_per_warp_main_1_map_2, -1, sizeof(int64_t) * main_1_map_2_cpw_size);
int64_t* d_cycles_per_warp_main_1_map_3;
auto main_1_map_3_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_map_3, sizeof(int64_t) * main_1_map_3_cpw_size);
hipMemset(d_cycles_per_warp_main_1_map_3, -1, sizeof(int64_t) * main_1_map_3_cpw_size);
int64_t* d_cycles_per_warp_main_1_aggregation_4;
auto main_1_aggregation_4_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_aggregation_4, sizeof(int64_t) * main_1_aggregation_4_cpw_size);
hipMemset(d_cycles_per_warp_main_1_aggregation_4, -1, sizeof(int64_t) * main_1_aggregation_4_cpw_size);
size_t COUNT4 = 5930889;
auto d_HT_4 = cuco::static_map{ (int)5930889*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_4;
hipMalloc(&d_SLOT_COUNT_4, sizeof(int));
hipMemset(d_SLOT_COUNT_4, 0, sizeof(int));
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr9;
hipMalloc(&d_aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT4);
hipMemset(d_aggr0__tmp_attr9, 0, sizeof(DBI64Type) * COUNT4);
DBDecimalType* d_aggr0__tmp_attr4;
hipMalloc(&d_aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT4);
hipMemset(d_aggr0__tmp_attr4, 0, sizeof(DBDecimalType) * COUNT4);
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT4);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT4);
DBDecimalType* d_aggr0__tmp_attr1;
hipMalloc(&d_aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT4);
hipMemset(d_aggr0__tmp_attr1, 0, sizeof(DBDecimalType) * COUNT4);
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT4);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT4);
DBDecimalType* d_aggr_rw__rw0;
hipMalloc(&d_aggr_rw__rw0, sizeof(DBDecimalType) * COUNT4);
hipMemset(d_aggr_rw__rw0, 0, sizeof(DBDecimalType) * COUNT4);
DBI64Type* d_aggr_rw__rw1;
hipMalloc(&d_aggr_rw__rw1, sizeof(DBI64Type) * COUNT4);
hipMemset(d_aggr_rw__rw1, 0, sizeof(DBI64Type) * COUNT4);
DBDecimalType* d_aggr_rw__rw2;
hipMalloc(&d_aggr_rw__rw2, sizeof(DBDecimalType) * COUNT4);
hipMemset(d_aggr_rw__rw2, 0, sizeof(DBDecimalType) * COUNT4);
DBI64Type* d_aggr_rw__rw3;
hipMalloc(&d_aggr_rw__rw3, sizeof(DBI64Type) * COUNT4);
hipMemset(d_aggr_rw__rw3, 0, sizeof(DBI64Type) * COUNT4);
DBDecimalType* d_aggr_rw__rw4;
hipMalloc(&d_aggr_rw__rw4, sizeof(DBDecimalType) * COUNT4);
hipMemset(d_aggr_rw__rw4, 0, sizeof(DBDecimalType) * COUNT4);
DBI64Type* d_aggr_rw__rw5;
hipMalloc(&d_aggr_rw__rw5, sizeof(DBI64Type) * COUNT4);
hipMemset(d_aggr_rw__rw5, 0, sizeof(DBI64Type) * COUNT4);
DBCharType* d_KEY_4lineitem__l_returnflag;
hipMalloc(&d_KEY_4lineitem__l_returnflag, sizeof(DBCharType) * COUNT4);
hipMemset(d_KEY_4lineitem__l_returnflag, 0, sizeof(DBCharType) * COUNT4);
DBCharType* d_KEY_4lineitem__l_linestatus;
hipMalloc(&d_KEY_4lineitem__l_linestatus, sizeof(DBCharType) * COUNT4);
hipMemset(d_KEY_4lineitem__l_linestatus, 0, sizeof(DBCharType) * COUNT4);
main_1<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TB>>>(d_HT_4.ref(cuco::insert_and_find), d_KEY_4lineitem__l_linestatus, d_KEY_4lineitem__l_returnflag, d_SLOT_COUNT_4, d_aggr0__tmp_attr0, d_aggr0__tmp_attr1, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_aggr0__tmp_attr9, d_aggr_rw__rw0, d_aggr_rw__rw1, d_aggr_rw__rw2, d_aggr_rw__rw3, d_aggr_rw__rw4, d_aggr_rw__rw5, d_cycles_per_warp_main_1_aggregation_4, d_cycles_per_warp_main_1_map_2, d_cycles_per_warp_main_1_map_3, d_cycles_per_warp_main_1_selection_0, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_linestatus, d_lineitem__l_quantity, d_lineitem__l_returnflag, d_lineitem__l_shipdate, d_lineitem__l_tax, lineitem_size);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_map_2 = (int64_t*)malloc(sizeof(int64_t) * main_1_map_2_cpw_size);
hipMemcpy(cycles_per_warp_main_1_map_2, d_cycles_per_warp_main_1_map_2, sizeof(int64_t) * main_1_map_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_map_2 ";
for (auto i=0ull; i < main_1_map_2_cpw_size; i++) std::cout << cycles_per_warp_main_1_map_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_map_3 = (int64_t*)malloc(sizeof(int64_t) * main_1_map_3_cpw_size);
hipMemcpy(cycles_per_warp_main_1_map_3, d_cycles_per_warp_main_1_map_3, sizeof(int64_t) * main_1_map_3_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_map_3 ";
for (auto i=0ull; i < main_1_map_3_cpw_size; i++) std::cout << cycles_per_warp_main_1_map_3[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_aggregation_4 = (int64_t*)malloc(sizeof(int64_t) * main_1_aggregation_4_cpw_size);
hipMemcpy(cycles_per_warp_main_1_aggregation_4, d_cycles_per_warp_main_1_aggregation_4, sizeof(int64_t) * main_1_aggregation_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_aggregation_4 ";
for (auto i=0ull; i < main_1_aggregation_4_cpw_size; i++) std::cout << cycles_per_warp_main_1_aggregation_4[i] << " ";
std::cout << std::endl;
COUNT4 = d_HT_4.size();
int64_t* d_cycles_per_warp_main_6_map_5;
auto main_6_map_5_cpw_size = std::ceil((float)COUNT4/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_6_map_5, sizeof(int64_t) * main_6_map_5_cpw_size);
hipMemset(d_cycles_per_warp_main_6_map_5, -1, sizeof(int64_t) * main_6_map_5_cpw_size);
int64_t* d_cycles_per_warp_main_6_map_7;
auto main_6_map_7_cpw_size = std::ceil((float)COUNT4/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_6_map_7, sizeof(int64_t) * main_6_map_7_cpw_size);
hipMemset(d_cycles_per_warp_main_6_map_7, -1, sizeof(int64_t) * main_6_map_7_cpw_size);
int64_t* d_cycles_per_warp_main_6_map_8;
auto main_6_map_8_cpw_size = std::ceil((float)COUNT4/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_6_map_8, sizeof(int64_t) * main_6_map_8_cpw_size);
hipMemset(d_cycles_per_warp_main_6_map_8, -1, sizeof(int64_t) * main_6_map_8_cpw_size);
int64_t* d_cycles_per_warp_main_6_materialize_9;
auto main_6_materialize_9_cpw_size = std::ceil((float)COUNT4/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_6_materialize_9, sizeof(int64_t) * main_6_materialize_9_cpw_size);
hipMemset(d_cycles_per_warp_main_6_materialize_9, -1, sizeof(int64_t) * main_6_materialize_9_cpw_size);
size_t COUNT9 = COUNT4;
//Materialize buffers
uint64_t* d_MAT_IDX9;
hipMalloc(&d_MAT_IDX9, sizeof(uint64_t));
hipMemset(d_MAT_IDX9, 0, sizeof(uint64_t));
auto MAT9lineitem__l_returnflag = (DBCharType*)malloc(sizeof(DBCharType) * COUNT9);
DBCharType* d_MAT9lineitem__l_returnflag;
hipMalloc(&d_MAT9lineitem__l_returnflag, sizeof(DBCharType) * COUNT9);
auto MAT9lineitem__l_linestatus = (DBCharType*)malloc(sizeof(DBCharType) * COUNT9);
DBCharType* d_MAT9lineitem__l_linestatus;
hipMalloc(&d_MAT9lineitem__l_linestatus, sizeof(DBCharType) * COUNT9);
auto MAT9aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT9);
DBDecimalType* d_MAT9aggr0__tmp_attr0;
hipMalloc(&d_MAT9aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT9);
auto MAT9aggr0__tmp_attr1 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT9);
DBDecimalType* d_MAT9aggr0__tmp_attr1;
hipMalloc(&d_MAT9aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT9);
auto MAT9aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT9);
DBDecimalType* d_MAT9aggr0__tmp_attr2;
hipMalloc(&d_MAT9aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT9);
auto MAT9aggr0__tmp_attr4 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT9);
DBDecimalType* d_MAT9aggr0__tmp_attr4;
hipMalloc(&d_MAT9aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT9);
auto MAT9aggr0__tmp_attr6 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT9);
DBDecimalType* d_MAT9aggr0__tmp_attr6;
hipMalloc(&d_MAT9aggr0__tmp_attr6, sizeof(DBDecimalType) * COUNT9);
auto MAT9aggr0__tmp_attr7 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT9);
DBDecimalType* d_MAT9aggr0__tmp_attr7;
hipMalloc(&d_MAT9aggr0__tmp_attr7, sizeof(DBDecimalType) * COUNT9);
auto MAT9aggr0__tmp_attr8 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT9);
DBDecimalType* d_MAT9aggr0__tmp_attr8;
hipMalloc(&d_MAT9aggr0__tmp_attr8, sizeof(DBDecimalType) * COUNT9);
auto MAT9aggr0__tmp_attr9 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT9);
DBI64Type* d_MAT9aggr0__tmp_attr9;
hipMalloc(&d_MAT9aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT9);
main_6<<<std::ceil((float)COUNT4/(float)TILE_SIZE), TB>>>(COUNT4, d_MAT9aggr0__tmp_attr0, d_MAT9aggr0__tmp_attr1, d_MAT9aggr0__tmp_attr2, d_MAT9aggr0__tmp_attr4, d_MAT9aggr0__tmp_attr6, d_MAT9aggr0__tmp_attr7, d_MAT9aggr0__tmp_attr8, d_MAT9aggr0__tmp_attr9, d_MAT9lineitem__l_linestatus, d_MAT9lineitem__l_returnflag, d_MAT_IDX9, d_aggr0__tmp_attr0, d_aggr0__tmp_attr1, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_aggr0__tmp_attr9, d_aggr_rw__rw0, d_aggr_rw__rw1, d_aggr_rw__rw2, d_aggr_rw__rw3, d_aggr_rw__rw4, d_aggr_rw__rw5, d_cycles_per_warp_main_6_map_5, d_cycles_per_warp_main_6_map_7, d_cycles_per_warp_main_6_map_8, d_cycles_per_warp_main_6_materialize_9, d_KEY_4lineitem__l_linestatus, d_KEY_4lineitem__l_returnflag);
uint64_t MATCOUNT_9 = 0;
hipMemcpy(&MATCOUNT_9, d_MAT_IDX9, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT9lineitem__l_returnflag, d_MAT9lineitem__l_returnflag, sizeof(DBCharType) * COUNT9, hipMemcpyDeviceToHost);
hipMemcpy(MAT9lineitem__l_linestatus, d_MAT9lineitem__l_linestatus, sizeof(DBCharType) * COUNT9, hipMemcpyDeviceToHost);
hipMemcpy(MAT9aggr0__tmp_attr0, d_MAT9aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT9, hipMemcpyDeviceToHost);
hipMemcpy(MAT9aggr0__tmp_attr1, d_MAT9aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT9, hipMemcpyDeviceToHost);
hipMemcpy(MAT9aggr0__tmp_attr2, d_MAT9aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT9, hipMemcpyDeviceToHost);
hipMemcpy(MAT9aggr0__tmp_attr4, d_MAT9aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT9, hipMemcpyDeviceToHost);
hipMemcpy(MAT9aggr0__tmp_attr6, d_MAT9aggr0__tmp_attr6, sizeof(DBDecimalType) * COUNT9, hipMemcpyDeviceToHost);
hipMemcpy(MAT9aggr0__tmp_attr7, d_MAT9aggr0__tmp_attr7, sizeof(DBDecimalType) * COUNT9, hipMemcpyDeviceToHost);
hipMemcpy(MAT9aggr0__tmp_attr8, d_MAT9aggr0__tmp_attr8, sizeof(DBDecimalType) * COUNT9, hipMemcpyDeviceToHost);
hipMemcpy(MAT9aggr0__tmp_attr9, d_MAT9aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT9, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_6_map_5 = (int64_t*)malloc(sizeof(int64_t) * main_6_map_5_cpw_size);
hipMemcpy(cycles_per_warp_main_6_map_5, d_cycles_per_warp_main_6_map_5, sizeof(int64_t) * main_6_map_5_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_6_map_5 ";
for (auto i=0ull; i < main_6_map_5_cpw_size; i++) std::cout << cycles_per_warp_main_6_map_5[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_6_map_7 = (int64_t*)malloc(sizeof(int64_t) * main_6_map_7_cpw_size);
hipMemcpy(cycles_per_warp_main_6_map_7, d_cycles_per_warp_main_6_map_7, sizeof(int64_t) * main_6_map_7_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_6_map_7 ";
for (auto i=0ull; i < main_6_map_7_cpw_size; i++) std::cout << cycles_per_warp_main_6_map_7[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_6_map_8 = (int64_t*)malloc(sizeof(int64_t) * main_6_map_8_cpw_size);
hipMemcpy(cycles_per_warp_main_6_map_8, d_cycles_per_warp_main_6_map_8, sizeof(int64_t) * main_6_map_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_6_map_8 ";
for (auto i=0ull; i < main_6_map_8_cpw_size; i++) std::cout << cycles_per_warp_main_6_map_8[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_6_materialize_9 = (int64_t*)malloc(sizeof(int64_t) * main_6_materialize_9_cpw_size);
hipMemcpy(cycles_per_warp_main_6_materialize_9, d_cycles_per_warp_main_6_materialize_9, sizeof(int64_t) * main_6_materialize_9_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_6_materialize_9 ";
for (auto i=0ull; i < main_6_materialize_9_cpw_size; i++) std::cout << cycles_per_warp_main_6_materialize_9[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_KEY_4lineitem__l_linestatus);
hipFree(d_KEY_4lineitem__l_returnflag);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr0__tmp_attr1);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_aggr0__tmp_attr4);
hipFree(d_aggr0__tmp_attr9);
hipFree(d_aggr_rw__rw0);
hipFree(d_aggr_rw__rw1);
hipFree(d_aggr_rw__rw2);
hipFree(d_aggr_rw__rw3);
hipFree(d_aggr_rw__rw4);
hipFree(d_aggr_rw__rw5);
hipFree(d_MAT9aggr0__tmp_attr0);
hipFree(d_MAT9aggr0__tmp_attr1);
hipFree(d_MAT9aggr0__tmp_attr2);
hipFree(d_MAT9aggr0__tmp_attr4);
hipFree(d_MAT9aggr0__tmp_attr6);
hipFree(d_MAT9aggr0__tmp_attr7);
hipFree(d_MAT9aggr0__tmp_attr8);
hipFree(d_MAT9aggr0__tmp_attr9);
hipFree(d_MAT9lineitem__l_linestatus);
hipFree(d_MAT9lineitem__l_returnflag);
hipFree(d_MAT_IDX9);
free(MAT9aggr0__tmp_attr0);
free(MAT9aggr0__tmp_attr1);
free(MAT9aggr0__tmp_attr2);
free(MAT9aggr0__tmp_attr4);
free(MAT9aggr0__tmp_attr6);
free(MAT9aggr0__tmp_attr7);
free(MAT9aggr0__tmp_attr8);
free(MAT9aggr0__tmp_attr9);
free(MAT9lineitem__l_linestatus);
free(MAT9lineitem__l_returnflag);
}