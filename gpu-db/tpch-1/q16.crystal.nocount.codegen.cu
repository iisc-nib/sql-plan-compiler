#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBStringType* part__p_brand, DBI32Type* part__p_partkey, DBI32Type* part__p_size, DBStringType* part__p_type, size_t part_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBI32Type reg_part__p_size[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_size[ITEM] = part__p_size[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_part__p_size[ITEM], 49, Predicate::eq)) || (evaluatePredicate(reg_part__p_size[ITEM], 14, Predicate::eq)) || (evaluatePredicate(reg_part__p_size[ITEM], 23, Predicate::eq)) || (evaluatePredicate(reg_part__p_size[ITEM], 45, Predicate::eq)) || (evaluatePredicate(reg_part__p_size[ITEM], 19, Predicate::eq)) || (evaluatePredicate(reg_part__p_size[ITEM], 3, Predicate::eq)) || (evaluatePredicate(reg_part__p_size[ITEM], 36, Predicate::eq)) || (evaluatePredicate(reg_part__p_size[ITEM], 9, Predicate::eq));
}
DBStringType reg_part__p_brand[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_brand[ITEM] = part__p_brand[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_part__p_brand[ITEM], "Brand#45", Predicate::neq);
}
DBStringType reg_part__p_type[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_type[ITEM] = part__p_type[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(Like(reg_part__p_type[ITEM], "MEDIUM POLISHED", "", nullptr, nullptr, 0));
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_0.insert(cuco::pair{KEY_0[ITEM], ITEM*TB + tid});
BUF_0[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_INSERT_SJ>
__global__ void main_3(HASHTABLE_INSERT_SJ HT_2, DBStringType* supplier__s_comment, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_supplier__s_comment[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_comment[ITEM] = supplier__s_comment[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= Like(reg_supplier__s_comment[ITEM], "", "", (const char*[]){ "Customer", "Complaints" }, (const int[]){ 8, 10 }, 2);
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_2.insert(cuco::pair{KEY_2[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_PROBE_SJ, typename HASHTABLE_FIND>
__global__ void main_5(uint64_t* BUF_0, HASHTABLE_PROBE HT_0, HASHTABLE_PROBE_SJ HT_2, HASHTABLE_FIND HT_4, DBI16Type* KEY_4part__p_brand_encoded, DBI32Type* KEY_4part__p_size, DBI16Type* KEY_4part__p_type_encoded, int* SLOT_COUNT_4, DBI64Type* aggr0__tmp_attr0, DBI16Type* part__p_brand_encoded, DBI32Type* part__p_size, DBI16Type* part__p_type_encoded, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_partsupp__ps_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
reg_partsupp__ps_partkey[ITEM] = partsupp__ps_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_partsupp__ps_partkey[ITEM];
}
//Probe Hash table
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_partsupp__ps_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
reg_partsupp__ps_suppkey[ITEM] = partsupp__ps_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_partsupp__ps_suppkey[ITEM];
}
//Probe Hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_2 = HT_2.find(KEY_2[ITEM]);
if (!(SLOT_2 == HT_2.end())) {selection_flags[ITEM] = 0;}
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI16Type reg_part__p_brand_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_brand_encoded[ITEM] = part__p_brand_encoded[BUF_0[slot_second0[ITEM] * 1 + 0]];
}
DBI16Type reg_part__p_type_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_type_encoded[ITEM] = part__p_type_encoded[BUF_0[slot_second0[ITEM] * 1 + 0]];
}
DBI32Type reg_part__p_size[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_size[ITEM] = part__p_size[BUF_0[slot_second0[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_part__p_brand_encoded[ITEM];
KEY_4[ITEM] <<= 16;
KEY_4[ITEM] |= reg_part__p_type_encoded[ITEM];
KEY_4[ITEM] <<= 32;
KEY_4[ITEM] |= reg_part__p_size[ITEM];
}
//Aggregate in hashtable
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_4 = get_aggregation_slot(KEY_4[ITEM], HT_4, SLOT_COUNT_4);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_4], 1);
KEY_4part__p_brand_encoded[buf_idx_4] = reg_part__p_brand_encoded[ITEM];
KEY_4part__p_type_encoded[buf_idx_4] = reg_part__p_type_encoded[ITEM];
KEY_4part__p_size[buf_idx_4] = reg_part__p_size[ITEM];
}
}
__global__ void main_7(size_t COUNT4, DBI64Type* MAT6aggr0__tmp_attr0, DBI16Type* MAT6part__p_brand_encoded, DBI32Type* MAT6part__p_size, DBI16Type* MAT6part__p_type_encoded, uint64_t* MAT_IDX6, DBI64Type* aggr0__tmp_attr0, DBI16Type* part__p_brand_encoded, DBI32Type* part__p_size, DBI16Type* part__p_type_encoded) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize buffers
DBI16Type reg_part__p_brand_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_part__p_brand_encoded[ITEM] = part__p_brand_encoded[ITEM*TB + tid];
}
DBI16Type reg_part__p_type_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_part__p_type_encoded[ITEM] = part__p_type_encoded[ITEM*TB + tid];
}
DBI32Type reg_part__p_size[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_part__p_size[ITEM] = part__p_size[ITEM*TB + tid];
}
DBI64Type reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx6 = atomicAdd((int*)MAT_IDX6, 1);
MAT6part__p_brand_encoded[mat_idx6] = reg_part__p_brand_encoded[ITEM];
MAT6part__p_type_encoded[mat_idx6] = reg_part__p_type_encoded[ITEM];
MAT6part__p_size[mat_idx6] = reg_part__p_size[ITEM];
MAT6aggr0__tmp_attr0[mat_idx6] = reg_aggr0__tmp_attr0[ITEM];
}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto start = std::chrono::high_resolution_clock::now();
size_t COUNT0 = part_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)part_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_part__p_brand, d_part__p_partkey, d_part__p_size, d_part__p_type, part_size);
size_t COUNT2 = supplier_size;
// Insert hash table control;
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_HT_2.ref(cuco::insert), d_supplier__s_comment, d_supplier__s_suppkey, supplier_size);
size_t COUNT4 = 120976;
auto d_HT_4 = cuco::static_map{ (int)120976*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_4;
hipMalloc(&d_SLOT_COUNT_4, sizeof(int));
hipMemset(d_SLOT_COUNT_4, 0, sizeof(int));
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT4);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT4);
DBI16Type* d_KEY_4part__p_brand_encoded;
hipMalloc(&d_KEY_4part__p_brand_encoded, sizeof(DBI16Type) * COUNT4);
hipMemset(d_KEY_4part__p_brand_encoded, 0, sizeof(DBI16Type) * COUNT4);
DBI16Type* d_KEY_4part__p_type_encoded;
hipMalloc(&d_KEY_4part__p_type_encoded, sizeof(DBI16Type) * COUNT4);
hipMemset(d_KEY_4part__p_type_encoded, 0, sizeof(DBI16Type) * COUNT4);
DBI32Type* d_KEY_4part__p_size;
hipMalloc(&d_KEY_4part__p_size, sizeof(DBI32Type) * COUNT4);
hipMemset(d_KEY_4part__p_size, 0, sizeof(DBI32Type) * COUNT4);
main_5<<<std::ceil((float)partsupp_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::insert_and_find), d_KEY_4part__p_brand_encoded, d_KEY_4part__p_size, d_KEY_4part__p_type_encoded, d_SLOT_COUNT_4, d_aggr0__tmp_attr0, d_part__p_brand_encoded, d_part__p_size, d_part__p_type_encoded, d_partsupp__ps_partkey, d_partsupp__ps_suppkey, partsupp_size);
COUNT4 = d_HT_4.size();
size_t COUNT6 = COUNT4;
//Materialize buffers
uint64_t* d_MAT_IDX6;
hipMalloc(&d_MAT_IDX6, sizeof(uint64_t));
hipMemset(d_MAT_IDX6, 0, sizeof(uint64_t));
auto MAT6part__p_brand_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT6);
DBI16Type* d_MAT6part__p_brand_encoded;
hipMalloc(&d_MAT6part__p_brand_encoded, sizeof(DBI16Type) * COUNT6);
auto MAT6part__p_type_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT6);
DBI16Type* d_MAT6part__p_type_encoded;
hipMalloc(&d_MAT6part__p_type_encoded, sizeof(DBI16Type) * COUNT6);
auto MAT6part__p_size = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT6);
DBI32Type* d_MAT6part__p_size;
hipMalloc(&d_MAT6part__p_size, sizeof(DBI32Type) * COUNT6);
auto MAT6aggr0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT6);
DBI64Type* d_MAT6aggr0__tmp_attr0;
hipMalloc(&d_MAT6aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT6);
main_7<<<std::ceil((float)COUNT4/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT4, d_MAT6aggr0__tmp_attr0, d_MAT6part__p_brand_encoded, d_MAT6part__p_size, d_MAT6part__p_type_encoded, d_MAT_IDX6, d_aggr0__tmp_attr0, d_KEY_4part__p_brand_encoded, d_KEY_4part__p_size, d_KEY_4part__p_type_encoded);
uint64_t MATCOUNT_6 = 0;
hipMemcpy(&MATCOUNT_6, d_MAT_IDX6, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT6part__p_brand_encoded, d_MAT6part__p_brand_encoded, sizeof(DBI16Type) * COUNT6, hipMemcpyDeviceToHost);
hipMemcpy(MAT6part__p_type_encoded, d_MAT6part__p_type_encoded, sizeof(DBI16Type) * COUNT6, hipMemcpyDeviceToHost);
hipMemcpy(MAT6part__p_size, d_MAT6part__p_size, sizeof(DBI32Type) * COUNT6, hipMemcpyDeviceToHost);
hipMemcpy(MAT6aggr0__tmp_attr0, d_MAT6aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT6, hipMemcpyDeviceToHost);
auto end = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < MATCOUNT_6; i++) { std::cout << "" << part__p_brand_map[MAT6part__p_brand_encoded[i]];
std::cout << "|" << part__p_type_map[MAT6part__p_type_encoded[i]];
std::cout << "|" << MAT6part__p_size[i];
std::cout << "|" << MAT6aggr0__tmp_attr0[i];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_KEY_4part__p_brand_encoded);
hipFree(d_KEY_4part__p_size);
hipFree(d_KEY_4part__p_type_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_MAT6aggr0__tmp_attr0);
hipFree(d_MAT6part__p_brand_encoded);
hipFree(d_MAT6part__p_size);
hipFree(d_MAT6part__p_type_encoded);
hipFree(d_MAT_IDX6);
free(MAT6aggr0__tmp_attr0);
free(MAT6part__p_brand_encoded);
free(MAT6part__p_size);
free(MAT6part__p_type_encoded);
}