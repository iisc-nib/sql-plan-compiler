#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, int64_t* cycles_per_warp_main_1_join_build_0, DBI32Type* nation__n_nationkey, size_t nation_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_nation__n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_nation__n_nationkey[ITEM] = nation__n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_nation__n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_0.insert(cuco::pair{KEY_0[ITEM], ITEM*TB + tid});
BUF_0[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_build_0[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_2(uint64_t* BUF_0, uint64_t* BUF_3, uint64_t* BUF_IDX_3, HASHTABLE_PROBE HT_0, HASHTABLE_INSERT HT_3, int64_t* cycles_per_warp_main_2_join_build_3, int64_t* cycles_per_warp_main_2_join_probe_0, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_nationkey[ITEM] = supplier__s_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_supplier__s_nationkey[ITEM];
}
//Probe Hash table
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_2_join_probe_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_3[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_3[ITEM] = 0;
KEY_3[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_3.insert(cuco::pair{KEY_3[ITEM], ITEM*TB + tid});
BUF_3[(ITEM*TB + tid) * 2 + 0] = ITEM*TB + tid;
BUF_3[(ITEM*TB + tid) * 2 + 1] = BUF_0[slot_second0[ITEM] * 1 + 0];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_2_join_build_3[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT>
__global__ void main_6(uint64_t* BUF_5, uint64_t* BUF_IDX_5, HASHTABLE_INSERT HT_5, int64_t* cycles_per_warp_main_6_join_build_5, DBI32Type* orders__o_orderkey, size_t orders_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_5[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_5[ITEM] = 0;
KEY_5[ITEM] |= reg_orders__o_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_5.insert(cuco::pair{KEY_5[ITEM], ITEM*TB + tid});
BUF_5[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_6_join_build_5[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT>
__global__ void main_9(uint64_t* BUF_8, uint64_t* BUF_IDX_8, HASHTABLE_INSERT HT_8, int64_t* cycles_per_warp_main_9_join_build_8, DBI32Type* part__p_partkey, size_t part_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_8.insert(cuco::pair{KEY_8[ITEM], ITEM*TB + tid});
BUF_8[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_9_join_build_8[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_4(uint64_t* BUF_10, uint64_t* BUF_3, uint64_t* BUF_IDX_10, HASHTABLE_INSERT HT_10, HASHTABLE_PROBE HT_3, int64_t* cycles_per_warp_main_4_join_build_10, int64_t* cycles_per_warp_main_4_join_probe_3, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size, DBI32Type* supplier__s_suppkey) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_3[ITEMS_PER_THREAD];
DBI32Type reg_partsupp__ps_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
reg_partsupp__ps_suppkey[ITEM] = partsupp__ps_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_3[ITEM] = 0;
KEY_3[ITEM] |= reg_partsupp__ps_suppkey[ITEM];
}
//Probe Hash table
int64_t slot_second3[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_3 = HT_3.find(KEY_3[ITEM]);
if (SLOT_3 == HT_3.end()) {selection_flags[ITEM] = 0; continue;}
slot_second3[ITEM] = SLOT_3->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_4_join_probe_3[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[BUF_3[slot_second3[ITEM] * 2 + 0]];
}
DBI32Type reg_partsupp__ps_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
reg_partsupp__ps_partkey[ITEM] = partsupp__ps_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_supplier__s_suppkey[ITEM];
KEY_10[ITEM] <<= 32;
KEY_10[ITEM] |= reg_partsupp__ps_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_10.insert(cuco::pair{KEY_10[ITEM], ITEM*TB + tid});
BUF_10[(ITEM*TB + tid) * 3 + 0] = BUF_3[slot_second3[ITEM] * 2 + 0];
BUF_10[(ITEM*TB + tid) * 3 + 1] = ITEM*TB + tid;
BUF_10[(ITEM*TB + tid) * 3 + 2] = BUF_3[slot_second3[ITEM] * 2 + 1];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_4_join_build_10[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_7(uint64_t* BUF_10, uint64_t* BUF_5, uint64_t* BUF_8, HASHTABLE_PROBE HT_10, HASHTABLE_FIND HT_13, HASHTABLE_PROBE HT_5, HASHTABLE_PROBE HT_8, DBI64Type* KEY_13map0__tmp_attr0, DBI16Type* KEY_13nation__n_name_encoded, int* SLOT_COUNT_13, DBDecimalType* aggr0__tmp_attr2, int64_t* cycles_per_warp_main_7_aggregation_13, int64_t* cycles_per_warp_main_7_join_probe_10, int64_t* cycles_per_warp_main_7_join_probe_5, int64_t* cycles_per_warp_main_7_join_probe_8, int64_t* cycles_per_warp_main_7_map_11, int64_t* cycles_per_warp_main_7_map_12, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded, DBDateType* orders__o_orderdate, DBDecimalType* partsupp__ps_supplycost) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_5[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_5[ITEM] = 0;
KEY_5[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
//Probe Hash table
int64_t slot_second5[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_5 = HT_5.find(KEY_5[ITEM]);
if (SLOT_5 == HT_5.end()) {selection_flags[ITEM] = 0; continue;}
slot_second5[ITEM] = SLOT_5->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_probe_5[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
//Probe Hash table
int64_t slot_second8[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_8 = HT_8.find(KEY_8[ITEM]);
if (SLOT_8 == HT_8.end()) {selection_flags[ITEM] = 0; continue;}
slot_second8[ITEM] = SLOT_8->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_probe_8[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_suppkey[ITEM] = lineitem__l_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_lineitem__l_suppkey[ITEM];
KEY_10[ITEM] <<= 32;
KEY_10[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
//Probe Hash table
int64_t slot_second10[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_10 = HT_10.find(KEY_10[ITEM]);
if (SLOT_10 == HT_10.end()) {selection_flags[ITEM] = 0; continue;}
slot_second10[ITEM] = SLOT_10->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_probe_10[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_map_11[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[BUF_5[slot_second5[ITEM] * 1 + 0]];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_map_12[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_13[ITEMS_PER_THREAD];
DBI16Type reg_nation__n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_nation__n_name_encoded[ITEM] = nation__n_name_encoded[BUF_10[slot_second10[ITEM] * 3 + 2]];
}
DBI64Type reg_map0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr0[ITEM] = ExtractFromDate("year", reg_orders__o_orderdate[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_13[ITEM] = 0;
KEY_13[ITEM] |= reg_nation__n_name_encoded[ITEM];
KEY_13[ITEM] <<= 32;
KEY_13[ITEM] |= (DBI32Type)reg_map0__tmp_attr0[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_quantity[ITEM] = lineitem__l_quantity[ITEM*TB + tid];
}
DBDecimalType reg_partsupp__ps_supplycost[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_partsupp__ps_supplycost[ITEM] = partsupp__ps_supplycost[BUF_10[slot_second10[ITEM] * 3 + 1]];
}
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = ((reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]))) - ((reg_partsupp__ps_supplycost[ITEM]) * (reg_lineitem__l_quantity[ITEM]));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_13 = get_aggregation_slot(KEY_13[ITEM], HT_13, SLOT_COUNT_13);
aggregate_sum(&aggr0__tmp_attr2[buf_idx_13], reg_map0__tmp_attr1[ITEM]);
KEY_13nation__n_name_encoded[buf_idx_13] = reg_nation__n_name_encoded[ITEM];
KEY_13map0__tmp_attr0[buf_idx_13] = reg_map0__tmp_attr0[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_aggregation_13[blockIdx.x] = cycles_per_warp;}
}
__global__ void main_15(size_t COUNT13, DBDecimalType* MAT14aggr0__tmp_attr2, DBI64Type* MAT14map0__tmp_attr0, DBI16Type* MAT14nation__n_name_encoded, uint64_t* MAT_IDX14, DBDecimalType* aggr0__tmp_attr2, int64_t* cycles_per_warp_main_15_materialize_14, DBI64Type* map0__tmp_attr0, DBI16Type* nation__n_name_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBI16Type reg_nation__n_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT13); ++ITEM) {
reg_nation__n_name_encoded[ITEM] = nation__n_name_encoded[ITEM*TB + tid];
}
DBI64Type reg_map0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT13); ++ITEM) {
reg_map0__tmp_attr0[ITEM] = map0__tmp_attr0[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT13); ++ITEM) {
reg_aggr0__tmp_attr2[ITEM] = aggr0__tmp_attr2[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT13); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx14 = atomicAdd((int*)MAT_IDX14, 1);
MAT14nation__n_name_encoded[mat_idx14] = reg_nation__n_name_encoded[ITEM];
MAT14map0__tmp_attr0[mat_idx14] = reg_map0__tmp_attr0[ITEM];
MAT14aggr0__tmp_attr2[mat_idx14] = reg_aggr0__tmp_attr2[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_15_materialize_14[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_1_join_build_0;
auto main_1_join_build_0_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_build_0, sizeof(int64_t) * main_1_join_build_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_build_0, -1, sizeof(int64_t) * main_1_join_build_0_cpw_size);
size_t COUNT0 = nation_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)nation_size/(float)TILE_SIZE), TB>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_cycles_per_warp_main_1_join_build_0, d_nation__n_nationkey, nation_size);
int64_t* cycles_per_warp_main_1_join_build_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_build_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_build_0, d_cycles_per_warp_main_1_join_build_0, sizeof(int64_t) * main_1_join_build_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_build_0 ";
for (auto i=0ull; i < main_1_join_build_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_build_0[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_2_join_probe_0;
auto main_2_join_probe_0_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_2_join_probe_0, sizeof(int64_t) * main_2_join_probe_0_cpw_size);
hipMemset(d_cycles_per_warp_main_2_join_probe_0, -1, sizeof(int64_t) * main_2_join_probe_0_cpw_size);
int64_t* d_cycles_per_warp_main_2_join_build_3;
auto main_2_join_build_3_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_2_join_build_3, sizeof(int64_t) * main_2_join_build_3_cpw_size);
hipMemset(d_cycles_per_warp_main_2_join_build_3, -1, sizeof(int64_t) * main_2_join_build_3_cpw_size);
size_t COUNT3 = supplier_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_3;
hipMalloc(&d_BUF_IDX_3, sizeof(uint64_t));
hipMemset(d_BUF_IDX_3, 0, sizeof(uint64_t));
uint64_t* d_BUF_3;
hipMalloc(&d_BUF_3, sizeof(uint64_t) * COUNT3 * 2);
auto d_HT_3 = cuco::static_map{ (int)COUNT3*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_2<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TB>>>(d_BUF_0, d_BUF_3, d_BUF_IDX_3, d_HT_0.ref(cuco::find), d_HT_3.ref(cuco::insert), d_cycles_per_warp_main_2_join_build_3, d_cycles_per_warp_main_2_join_probe_0, d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
int64_t* cycles_per_warp_main_2_join_probe_0 = (int64_t*)malloc(sizeof(int64_t) * main_2_join_probe_0_cpw_size);
hipMemcpy(cycles_per_warp_main_2_join_probe_0, d_cycles_per_warp_main_2_join_probe_0, sizeof(int64_t) * main_2_join_probe_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_2_join_probe_0 ";
for (auto i=0ull; i < main_2_join_probe_0_cpw_size; i++) std::cout << cycles_per_warp_main_2_join_probe_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_2_join_build_3 = (int64_t*)malloc(sizeof(int64_t) * main_2_join_build_3_cpw_size);
hipMemcpy(cycles_per_warp_main_2_join_build_3, d_cycles_per_warp_main_2_join_build_3, sizeof(int64_t) * main_2_join_build_3_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_2_join_build_3 ";
for (auto i=0ull; i < main_2_join_build_3_cpw_size; i++) std::cout << cycles_per_warp_main_2_join_build_3[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_6_join_build_5;
auto main_6_join_build_5_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_6_join_build_5, sizeof(int64_t) * main_6_join_build_5_cpw_size);
hipMemset(d_cycles_per_warp_main_6_join_build_5, -1, sizeof(int64_t) * main_6_join_build_5_cpw_size);
size_t COUNT5 = orders_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_5;
hipMalloc(&d_BUF_IDX_5, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5, 0, sizeof(uint64_t));
uint64_t* d_BUF_5;
hipMalloc(&d_BUF_5, sizeof(uint64_t) * COUNT5 * 1);
auto d_HT_5 = cuco::static_map{ (int)COUNT5*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_6<<<std::ceil((float)orders_size/(float)TILE_SIZE), TB>>>(d_BUF_5, d_BUF_IDX_5, d_HT_5.ref(cuco::insert), d_cycles_per_warp_main_6_join_build_5, d_orders__o_orderkey, orders_size);
int64_t* cycles_per_warp_main_6_join_build_5 = (int64_t*)malloc(sizeof(int64_t) * main_6_join_build_5_cpw_size);
hipMemcpy(cycles_per_warp_main_6_join_build_5, d_cycles_per_warp_main_6_join_build_5, sizeof(int64_t) * main_6_join_build_5_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_6_join_build_5 ";
for (auto i=0ull; i < main_6_join_build_5_cpw_size; i++) std::cout << cycles_per_warp_main_6_join_build_5[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_9_join_build_8;
auto main_9_join_build_8_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_9_join_build_8, sizeof(int64_t) * main_9_join_build_8_cpw_size);
hipMemset(d_cycles_per_warp_main_9_join_build_8, -1, sizeof(int64_t) * main_9_join_build_8_cpw_size);
size_t COUNT8 = part_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_8;
hipMalloc(&d_BUF_IDX_8, sizeof(uint64_t));
hipMemset(d_BUF_IDX_8, 0, sizeof(uint64_t));
uint64_t* d_BUF_8;
hipMalloc(&d_BUF_8, sizeof(uint64_t) * COUNT8 * 1);
auto d_HT_8 = cuco::static_map{ (int)COUNT8*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_9<<<std::ceil((float)part_size/(float)TILE_SIZE), TB>>>(d_BUF_8, d_BUF_IDX_8, d_HT_8.ref(cuco::insert), d_cycles_per_warp_main_9_join_build_8, d_part__p_partkey, part_size);
int64_t* cycles_per_warp_main_9_join_build_8 = (int64_t*)malloc(sizeof(int64_t) * main_9_join_build_8_cpw_size);
hipMemcpy(cycles_per_warp_main_9_join_build_8, d_cycles_per_warp_main_9_join_build_8, sizeof(int64_t) * main_9_join_build_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_9_join_build_8 ";
for (auto i=0ull; i < main_9_join_build_8_cpw_size; i++) std::cout << cycles_per_warp_main_9_join_build_8[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_4_join_probe_3;
auto main_4_join_probe_3_cpw_size = std::ceil((float)partsupp_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_4_join_probe_3, sizeof(int64_t) * main_4_join_probe_3_cpw_size);
hipMemset(d_cycles_per_warp_main_4_join_probe_3, -1, sizeof(int64_t) * main_4_join_probe_3_cpw_size);
int64_t* d_cycles_per_warp_main_4_join_build_10;
auto main_4_join_build_10_cpw_size = std::ceil((float)partsupp_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_4_join_build_10, sizeof(int64_t) * main_4_join_build_10_cpw_size);
hipMemset(d_cycles_per_warp_main_4_join_build_10, -1, sizeof(int64_t) * main_4_join_build_10_cpw_size);
size_t COUNT10 = partsupp_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_10;
hipMalloc(&d_BUF_IDX_10, sizeof(uint64_t));
hipMemset(d_BUF_IDX_10, 0, sizeof(uint64_t));
uint64_t* d_BUF_10;
hipMalloc(&d_BUF_10, sizeof(uint64_t) * COUNT10 * 3);
auto d_HT_10 = cuco::static_map{ (int)COUNT10*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_4<<<std::ceil((float)partsupp_size/(float)TILE_SIZE), TB>>>(d_BUF_10, d_BUF_3, d_BUF_IDX_10, d_HT_10.ref(cuco::insert), d_HT_3.ref(cuco::find), d_cycles_per_warp_main_4_join_build_10, d_cycles_per_warp_main_4_join_probe_3, d_partsupp__ps_partkey, d_partsupp__ps_suppkey, partsupp_size, d_supplier__s_suppkey);
int64_t* cycles_per_warp_main_4_join_probe_3 = (int64_t*)malloc(sizeof(int64_t) * main_4_join_probe_3_cpw_size);
hipMemcpy(cycles_per_warp_main_4_join_probe_3, d_cycles_per_warp_main_4_join_probe_3, sizeof(int64_t) * main_4_join_probe_3_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_4_join_probe_3 ";
for (auto i=0ull; i < main_4_join_probe_3_cpw_size; i++) std::cout << cycles_per_warp_main_4_join_probe_3[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_4_join_build_10 = (int64_t*)malloc(sizeof(int64_t) * main_4_join_build_10_cpw_size);
hipMemcpy(cycles_per_warp_main_4_join_build_10, d_cycles_per_warp_main_4_join_build_10, sizeof(int64_t) * main_4_join_build_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_4_join_build_10 ";
for (auto i=0ull; i < main_4_join_build_10_cpw_size; i++) std::cout << cycles_per_warp_main_4_join_build_10[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_7_join_probe_5;
auto main_7_join_probe_5_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_probe_5, sizeof(int64_t) * main_7_join_probe_5_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_probe_5, -1, sizeof(int64_t) * main_7_join_probe_5_cpw_size);
int64_t* d_cycles_per_warp_main_7_join_probe_8;
auto main_7_join_probe_8_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_probe_8, sizeof(int64_t) * main_7_join_probe_8_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_probe_8, -1, sizeof(int64_t) * main_7_join_probe_8_cpw_size);
int64_t* d_cycles_per_warp_main_7_join_probe_10;
auto main_7_join_probe_10_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_probe_10, sizeof(int64_t) * main_7_join_probe_10_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_probe_10, -1, sizeof(int64_t) * main_7_join_probe_10_cpw_size);
int64_t* d_cycles_per_warp_main_7_map_11;
auto main_7_map_11_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_map_11, sizeof(int64_t) * main_7_map_11_cpw_size);
hipMemset(d_cycles_per_warp_main_7_map_11, -1, sizeof(int64_t) * main_7_map_11_cpw_size);
int64_t* d_cycles_per_warp_main_7_map_12;
auto main_7_map_12_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_map_12, sizeof(int64_t) * main_7_map_12_cpw_size);
hipMemset(d_cycles_per_warp_main_7_map_12, -1, sizeof(int64_t) * main_7_map_12_cpw_size);
int64_t* d_cycles_per_warp_main_7_aggregation_13;
auto main_7_aggregation_13_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_aggregation_13, sizeof(int64_t) * main_7_aggregation_13_cpw_size);
hipMemset(d_cycles_per_warp_main_7_aggregation_13, -1, sizeof(int64_t) * main_7_aggregation_13_cpw_size);
size_t COUNT13 = 48009721;
auto d_HT_13 = cuco::static_map{ (int)48009721*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_13;
hipMalloc(&d_SLOT_COUNT_13, sizeof(int));
hipMemset(d_SLOT_COUNT_13, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT13);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT13);
DBI16Type* d_KEY_13nation__n_name_encoded;
hipMalloc(&d_KEY_13nation__n_name_encoded, sizeof(DBI16Type) * COUNT13);
hipMemset(d_KEY_13nation__n_name_encoded, 0, sizeof(DBI16Type) * COUNT13);
DBI64Type* d_KEY_13map0__tmp_attr0;
hipMalloc(&d_KEY_13map0__tmp_attr0, sizeof(DBI64Type) * COUNT13);
hipMemset(d_KEY_13map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT13);
main_7<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TB>>>(d_BUF_10, d_BUF_5, d_BUF_8, d_HT_10.ref(cuco::find), d_HT_13.ref(cuco::insert_and_find), d_HT_5.ref(cuco::find), d_HT_8.ref(cuco::find), d_KEY_13map0__tmp_attr0, d_KEY_13nation__n_name_encoded, d_SLOT_COUNT_13, d_aggr0__tmp_attr2, d_cycles_per_warp_main_7_aggregation_13, d_cycles_per_warp_main_7_join_probe_10, d_cycles_per_warp_main_7_join_probe_5, d_cycles_per_warp_main_7_join_probe_8, d_cycles_per_warp_main_7_map_11, d_cycles_per_warp_main_7_map_12, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_quantity, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded, d_orders__o_orderdate, d_partsupp__ps_supplycost);
int64_t* cycles_per_warp_main_7_join_probe_5 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_probe_5_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_probe_5, d_cycles_per_warp_main_7_join_probe_5, sizeof(int64_t) * main_7_join_probe_5_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_probe_5 ";
for (auto i=0ull; i < main_7_join_probe_5_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_probe_5[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_join_probe_8 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_probe_8_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_probe_8, d_cycles_per_warp_main_7_join_probe_8, sizeof(int64_t) * main_7_join_probe_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_probe_8 ";
for (auto i=0ull; i < main_7_join_probe_8_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_probe_8[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_join_probe_10 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_probe_10_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_probe_10, d_cycles_per_warp_main_7_join_probe_10, sizeof(int64_t) * main_7_join_probe_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_probe_10 ";
for (auto i=0ull; i < main_7_join_probe_10_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_probe_10[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_map_11 = (int64_t*)malloc(sizeof(int64_t) * main_7_map_11_cpw_size);
hipMemcpy(cycles_per_warp_main_7_map_11, d_cycles_per_warp_main_7_map_11, sizeof(int64_t) * main_7_map_11_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_map_11 ";
for (auto i=0ull; i < main_7_map_11_cpw_size; i++) std::cout << cycles_per_warp_main_7_map_11[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_map_12 = (int64_t*)malloc(sizeof(int64_t) * main_7_map_12_cpw_size);
hipMemcpy(cycles_per_warp_main_7_map_12, d_cycles_per_warp_main_7_map_12, sizeof(int64_t) * main_7_map_12_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_map_12 ";
for (auto i=0ull; i < main_7_map_12_cpw_size; i++) std::cout << cycles_per_warp_main_7_map_12[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_aggregation_13 = (int64_t*)malloc(sizeof(int64_t) * main_7_aggregation_13_cpw_size);
hipMemcpy(cycles_per_warp_main_7_aggregation_13, d_cycles_per_warp_main_7_aggregation_13, sizeof(int64_t) * main_7_aggregation_13_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_aggregation_13 ";
for (auto i=0ull; i < main_7_aggregation_13_cpw_size; i++) std::cout << cycles_per_warp_main_7_aggregation_13[i] << " ";
std::cout << std::endl;
COUNT13 = d_HT_13.size();
int64_t* d_cycles_per_warp_main_15_materialize_14;
auto main_15_materialize_14_cpw_size = std::ceil((float)COUNT13/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_15_materialize_14, sizeof(int64_t) * main_15_materialize_14_cpw_size);
hipMemset(d_cycles_per_warp_main_15_materialize_14, -1, sizeof(int64_t) * main_15_materialize_14_cpw_size);
size_t COUNT14 = COUNT13;
//Materialize buffers
uint64_t* d_MAT_IDX14;
hipMalloc(&d_MAT_IDX14, sizeof(uint64_t));
hipMemset(d_MAT_IDX14, 0, sizeof(uint64_t));
auto MAT14nation__n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT14);
DBI16Type* d_MAT14nation__n_name_encoded;
hipMalloc(&d_MAT14nation__n_name_encoded, sizeof(DBI16Type) * COUNT14);
auto MAT14map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT14);
DBI64Type* d_MAT14map0__tmp_attr0;
hipMalloc(&d_MAT14map0__tmp_attr0, sizeof(DBI64Type) * COUNT14);
auto MAT14aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT14);
DBDecimalType* d_MAT14aggr0__tmp_attr2;
hipMalloc(&d_MAT14aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT14);
main_15<<<std::ceil((float)COUNT13/(float)TILE_SIZE), TB>>>(COUNT13, d_MAT14aggr0__tmp_attr2, d_MAT14map0__tmp_attr0, d_MAT14nation__n_name_encoded, d_MAT_IDX14, d_aggr0__tmp_attr2, d_cycles_per_warp_main_15_materialize_14, d_KEY_13map0__tmp_attr0, d_KEY_13nation__n_name_encoded);
uint64_t MATCOUNT_14 = 0;
hipMemcpy(&MATCOUNT_14, d_MAT_IDX14, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT14nation__n_name_encoded, d_MAT14nation__n_name_encoded, sizeof(DBI16Type) * COUNT14, hipMemcpyDeviceToHost);
hipMemcpy(MAT14map0__tmp_attr0, d_MAT14map0__tmp_attr0, sizeof(DBI64Type) * COUNT14, hipMemcpyDeviceToHost);
hipMemcpy(MAT14aggr0__tmp_attr2, d_MAT14aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT14, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_15_materialize_14 = (int64_t*)malloc(sizeof(int64_t) * main_15_materialize_14_cpw_size);
hipMemcpy(cycles_per_warp_main_15_materialize_14, d_cycles_per_warp_main_15_materialize_14, sizeof(int64_t) * main_15_materialize_14_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_15_materialize_14 ";
for (auto i=0ull; i < main_15_materialize_14_cpw_size; i++) std::cout << cycles_per_warp_main_15_materialize_14[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_BUF_3);
hipFree(d_BUF_IDX_3);
hipFree(d_BUF_5);
hipFree(d_BUF_IDX_5);
hipFree(d_BUF_8);
hipFree(d_BUF_IDX_8);
hipFree(d_BUF_10);
hipFree(d_BUF_IDX_10);
hipFree(d_KEY_13map0__tmp_attr0);
hipFree(d_KEY_13nation__n_name_encoded);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_MAT14aggr0__tmp_attr2);
hipFree(d_MAT14map0__tmp_attr0);
hipFree(d_MAT14nation__n_name_encoded);
hipFree(d_MAT_IDX14);
free(MAT14aggr0__tmp_attr2);
free(MAT14map0__tmp_attr0);
free(MAT14nation__n_name_encoded);
}