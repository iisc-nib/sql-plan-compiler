#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_0.insert(cuco::pair{KEY_0[ITEM], ITEM*TB + tid});
BUF_0[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_3(uint64_t* BUF_0, HASHTABLE_PROBE HT_0, HASHTABLE_FIND HT_2, DBI32Type* KEY_2customer__c_custkey, int* SLOT_COUNT_2, DBI64Type* aggr0__tmp_attr0, DBI32Type* customer__c_custkey, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_custkey[ITEM] = orders__o_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_orders__o_custkey[ITEM];
}
//Probe Hash table
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_custkey[ITEM] = customer__c_custkey[BUF_0[slot_second0[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_customer__c_custkey[ITEM];
}
//Aggregate in hashtable
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_2 = get_aggregation_slot(KEY_2[ITEM], HT_2, SLOT_COUNT_2);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_2], 1);
KEY_2customer__c_custkey[buf_idx_2] = reg_customer__c_custkey[ITEM];
}
}
template<typename HASHTABLE_FIND>
__global__ void main_5(size_t COUNT2, HASHTABLE_FIND HT_4, DBI64Type* KEY_4aggr0__tmp_attr0, int* SLOT_COUNT_4, DBI64Type* aggr0__tmp_attr0, DBI64Type* aggr1__tmp_attr1) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI64Type reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= (DBI32Type)reg_aggr0__tmp_attr0[ITEM];
}
//Aggregate in hashtable
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_4 = get_aggregation_slot(KEY_4[ITEM], HT_4, SLOT_COUNT_4);
aggregate_sum(&aggr1__tmp_attr1[buf_idx_4], 1);
KEY_4aggr0__tmp_attr0[buf_idx_4] = reg_aggr0__tmp_attr0[ITEM];
}
}
__global__ void main_7(size_t COUNT4, DBI64Type* MAT6aggr0__tmp_attr0, DBI64Type* MAT6aggr1__tmp_attr1, uint64_t* MAT_IDX6, DBI64Type* aggr0__tmp_attr0, DBI64Type* aggr1__tmp_attr1) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize buffers
DBI64Type reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
DBI64Type reg_aggr1__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr1__tmp_attr1[ITEM] = aggr1__tmp_attr1[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx6 = atomicAdd((int*)MAT_IDX6, 1);
MAT6aggr0__tmp_attr0[mat_idx6] = reg_aggr0__tmp_attr0[ITEM];
MAT6aggr1__tmp_attr1[mat_idx6] = reg_aggr1__tmp_attr1[ITEM];
}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto start = std::chrono::high_resolution_clock::now();
size_t COUNT0 = customer_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)customer_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_customer__c_custkey, customer_size);
size_t COUNT2 = 1500000;
auto d_HT_2 = cuco::static_map{ (int)1500000*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_2;
hipMalloc(&d_SLOT_COUNT_2, sizeof(int));
hipMemset(d_SLOT_COUNT_2, 0, sizeof(int));
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT2);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT2);
DBI32Type* d_KEY_2customer__c_custkey;
hipMalloc(&d_KEY_2customer__c_custkey, sizeof(DBI32Type) * COUNT2);
hipMemset(d_KEY_2customer__c_custkey, 0, sizeof(DBI32Type) * COUNT2);
main_3<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::insert_and_find), d_KEY_2customer__c_custkey, d_SLOT_COUNT_2, d_aggr0__tmp_attr0, d_customer__c_custkey, d_orders__o_custkey, d_orders__o_orderkey, orders_size);
COUNT2 = d_HT_2.size();
size_t COUNT4 = 1500000;
auto d_HT_4 = cuco::static_map{ (int)1500000*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_4;
hipMalloc(&d_SLOT_COUNT_4, sizeof(int));
hipMemset(d_SLOT_COUNT_4, 0, sizeof(int));
//Aggregate in hashtable
DBI64Type* d_aggr1__tmp_attr1;
hipMalloc(&d_aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT4);
hipMemset(d_aggr1__tmp_attr1, 0, sizeof(DBI64Type) * COUNT4);
DBI64Type* d_KEY_4aggr0__tmp_attr0;
hipMalloc(&d_KEY_4aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT4);
hipMemset(d_KEY_4aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT4);
main_5<<<std::ceil((float)COUNT2/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT2, d_HT_4.ref(cuco::insert_and_find), d_KEY_4aggr0__tmp_attr0, d_SLOT_COUNT_4, d_aggr0__tmp_attr0, d_aggr1__tmp_attr1);
COUNT4 = d_HT_4.size();
size_t COUNT6 = COUNT4;
//Materialize buffers
uint64_t* d_MAT_IDX6;
hipMalloc(&d_MAT_IDX6, sizeof(uint64_t));
hipMemset(d_MAT_IDX6, 0, sizeof(uint64_t));
auto MAT6aggr0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT6);
DBI64Type* d_MAT6aggr0__tmp_attr0;
hipMalloc(&d_MAT6aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT6);
auto MAT6aggr1__tmp_attr1 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT6);
DBI64Type* d_MAT6aggr1__tmp_attr1;
hipMalloc(&d_MAT6aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT6);
main_7<<<std::ceil((float)COUNT4/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT4, d_MAT6aggr0__tmp_attr0, d_MAT6aggr1__tmp_attr1, d_MAT_IDX6, d_KEY_4aggr0__tmp_attr0, d_aggr1__tmp_attr1);
uint64_t MATCOUNT_6 = 0;
hipMemcpy(&MATCOUNT_6, d_MAT_IDX6, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT6aggr0__tmp_attr0, d_MAT6aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT6, hipMemcpyDeviceToHost);
hipMemcpy(MAT6aggr1__tmp_attr1, d_MAT6aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT6, hipMemcpyDeviceToHost);
auto end = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < MATCOUNT_6; i++) { std::cout << "" << MAT6aggr0__tmp_attr0[i];
std::cout << "|" << MAT6aggr1__tmp_attr1[i];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_KEY_2customer__c_custkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_KEY_4aggr0__tmp_attr0);
hipFree(d_aggr1__tmp_attr1);
hipFree(d_MAT6aggr0__tmp_attr0);
hipFree(d_MAT6aggr1__tmp_attr1);
hipFree(d_MAT_IDX6);
free(MAT6aggr0__tmp_attr0);
free(MAT6aggr1__tmp_attr1);
}