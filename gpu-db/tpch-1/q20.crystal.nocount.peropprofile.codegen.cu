#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_INSERT_SJ>
__global__ void main_3(HASHTABLE_INSERT_SJ HT_6, int64_t* cycles_per_warp_main_3_selection_2, int64_t* cycles_per_warp_main_3_semi_join_build_6, DBStringType* part__p_name, DBI32Type* part__p_partkey, size_t part_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_part__p_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_name[ITEM] = part__p_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= Like(reg_part__p_name[ITEM], "forest", "", nullptr, nullptr, 0);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_6.insert(cuco::pair{KEY_6[ITEM], 1});
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_3_semi_join_build_6[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE_SJ, typename HASHTABLE_INSERT>
__global__ void main_7(uint64_t* BUF_8, uint64_t* BUF_IDX_8, HASHTABLE_PROBE_SJ HT_6, HASHTABLE_INSERT HT_8, int64_t* cycles_per_warp_main_7_join_build_8, int64_t* cycles_per_warp_main_7_semi_join_probe_6, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_partsupp__ps_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
reg_partsupp__ps_partkey[ITEM] = partsupp__ps_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_partsupp__ps_partkey[ITEM];
}
//Probe Hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0;}
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_semi_join_probe_6[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_partsupp__ps_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
reg_partsupp__ps_suppkey[ITEM] = partsupp__ps_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_partsupp__ps_partkey[ITEM];
KEY_8[ITEM] <<= 32;
KEY_8[ITEM] |= reg_partsupp__ps_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_8.insert(cuco::pair{KEY_8[ITEM], ITEM*TB + tid});
BUF_8[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_7_join_build_8[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_5(uint64_t* BUF_8, HASHTABLE_PROBE HT_8, HASHTABLE_FIND HT_9, DBI32Type* KEY_9lineitem__l_partkey, DBI32Type* KEY_9lineitem__l_suppkey, int* SLOT_COUNT_9, DBDecimalType* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_5_aggregation_9, int64_t* cycles_per_warp_main_5_join_probe_8, int64_t* cycles_per_warp_main_5_selection_4, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI32Type* moved_aggr__ps_suppkey, DBI32Type* moved_aggr_u_1__ps_availqty, DBI32Type* moved_aggr_u_2__ps_partkey, DBI32Type* partsupp__ps_availqty, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBDateType reg_lineitem__l_shipdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipdate[ITEM] = lineitem__l_shipdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 9131, Predicate::lt);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_selection_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
DBI32Type reg_lineitem__l_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_suppkey[ITEM] = lineitem__l_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_lineitem__l_partkey[ITEM];
KEY_8[ITEM] <<= 32;
KEY_8[ITEM] |= reg_lineitem__l_suppkey[ITEM];
}
//Probe Hash table
int64_t slot_second8[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_8 = HT_8.find(KEY_8[ITEM]);
if (SLOT_8 == HT_8.end()) {selection_flags[ITEM] = 0; continue;}
slot_second8[ITEM] = SLOT_8->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_join_probe_8[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_9[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_9[ITEM] = 0;
KEY_9[ITEM] |= reg_lineitem__l_suppkey[ITEM];
KEY_9[ITEM] <<= 32;
KEY_9[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_quantity[ITEM] = lineitem__l_quantity[ITEM*TB + tid];
}
DBI32Type reg_partsupp__ps_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_partsupp__ps_suppkey[ITEM] = partsupp__ps_suppkey[BUF_8[slot_second8[ITEM] * 1 + 0]];
}
DBI32Type reg_partsupp__ps_availqty[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_partsupp__ps_availqty[ITEM] = partsupp__ps_availqty[BUF_8[slot_second8[ITEM] * 1 + 0]];
}
DBI32Type reg_partsupp__ps_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_partsupp__ps_partkey[ITEM] = partsupp__ps_partkey[BUF_8[slot_second8[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_9 = get_aggregation_slot(KEY_9[ITEM], HT_9, SLOT_COUNT_9);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_9], reg_lineitem__l_quantity[ITEM]);
aggregate_any(&moved_aggr__ps_suppkey[buf_idx_9], reg_partsupp__ps_suppkey[ITEM]);
aggregate_any(&moved_aggr_u_1__ps_availqty[buf_idx_9], reg_partsupp__ps_availqty[ITEM]);
aggregate_any(&moved_aggr_u_2__ps_partkey[buf_idx_9], reg_partsupp__ps_partkey[ITEM]);
KEY_9lineitem__l_suppkey[buf_idx_9] = reg_lineitem__l_suppkey[ITEM];
KEY_9lineitem__l_partkey[buf_idx_9] = reg_lineitem__l_partkey[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_5_aggregation_9[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_10, uint64_t* BUF_IDX_10, HASHTABLE_INSERT HT_10, int64_t* cycles_per_warp_main_1_join_build_10, int64_t* cycles_per_warp_main_1_selection_0, DBStringType* nation__n_name, DBI32Type* nation__n_nationkey, size_t nation_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBStringType reg_nation__n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_nation__n_name[ITEM] = nation__n_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_nation__n_name[ITEM], "CANADA", Predicate::eq);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI32Type reg_nation__n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_nation__n_nationkey[ITEM] = nation__n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_nation__n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_10.insert(cuco::pair{KEY_10[ITEM], ITEM*TB + tid});
BUF_10[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_build_10[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_INSERT_SJ>
__global__ void main_13(size_t COUNT9, HASHTABLE_INSERT_SJ HT_16, DBDecimalType* aggr0__tmp_attr0, int64_t* cycles_per_warp_main_13_map_12, int64_t* cycles_per_warp_main_13_selection_14, int64_t* cycles_per_warp_main_13_selection_15, int64_t* cycles_per_warp_main_13_semi_join_build_16, DBI32Type* moved_aggr__ps_suppkey, DBI32Type* moved_aggr_u_1__ps_availqty, DBI32Type* moved_aggr_u_2__ps_partkey) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_13_map_12[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_13_selection_14[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
DBI32Type reg_partsupp__ps_availqty[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
reg_partsupp__ps_availqty[ITEM] = moved_aggr_u_1__ps_availqty[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (0.5) * (reg_aggr0__tmp_attr0[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((true) && (evaluatePredicate(((DBDecimalType)reg_partsupp__ps_availqty[ITEM]), reg_map0__tmp_attr1[ITEM], Predicate::gt))) && (true);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_13_selection_15[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_16[ITEMS_PER_THREAD];
DBI32Type reg_partsupp__ps_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
reg_partsupp__ps_suppkey[ITEM] = moved_aggr__ps_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_16[ITEM] = 0;
KEY_16[ITEM] |= reg_partsupp__ps_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_16.insert(cuco::pair{KEY_16[ITEM], 1});
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_13_semi_join_build_16[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_PROBE_SJ>
__global__ void main_11(uint64_t* BUF_10, HASHTABLE_PROBE HT_10, HASHTABLE_PROBE_SJ HT_16, DBI16Type* MAT17supplier__s_address_encoded, DBI16Type* MAT17supplier__s_name_encoded, uint64_t* MAT_IDX17, int64_t* cycles_per_warp_main_11_join_probe_10, int64_t* cycles_per_warp_main_11_materialize_17, int64_t* cycles_per_warp_main_11_semi_join_probe_16, DBI16Type* supplier__s_address_encoded, DBI16Type* supplier__s_name_encoded, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_nationkey[ITEM] = supplier__s_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_supplier__s_nationkey[ITEM];
}
//Probe Hash table
int64_t slot_second10[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_10 = HT_10.find(KEY_10[ITEM]);
if (SLOT_10 == HT_10.end()) {selection_flags[ITEM] = 0; continue;}
slot_second10[ITEM] = SLOT_10->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_join_probe_10[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_16[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_16[ITEM] = 0;
KEY_16[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
//Probe Hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_16 = HT_16.find(KEY_16[ITEM]);
if (SLOT_16 == HT_16.end()) {selection_flags[ITEM] = 0;}
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_semi_join_probe_16[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBI16Type reg_supplier__s_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_name_encoded[ITEM] = supplier__s_name_encoded[ITEM*TB + tid];
}
DBI16Type reg_supplier__s_address_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_address_encoded[ITEM] = supplier__s_address_encoded[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx17 = atomicAdd((int*)MAT_IDX17, 1);
MAT17supplier__s_name_encoded[mat_idx17] = reg_supplier__s_name_encoded[ITEM];
MAT17supplier__s_address_encoded[mat_idx17] = reg_supplier__s_address_encoded[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_materialize_17[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
int64_t* d_cycles_per_warp_main_3_selection_2;
auto main_3_selection_2_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_selection_2, sizeof(int64_t) * main_3_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_3_selection_2, -1, sizeof(int64_t) * main_3_selection_2_cpw_size);
int64_t* d_cycles_per_warp_main_3_semi_join_build_6;
auto main_3_semi_join_build_6_cpw_size = std::ceil((float)part_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_3_semi_join_build_6, sizeof(int64_t) * main_3_semi_join_build_6_cpw_size);
hipMemset(d_cycles_per_warp_main_3_semi_join_build_6, -1, sizeof(int64_t) * main_3_semi_join_build_6_cpw_size);
size_t COUNT6 = part_size;
// Insert hash table control;
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)part_size/(float)TILE_SIZE), TB>>>(d_HT_6.ref(cuco::insert), d_cycles_per_warp_main_3_selection_2, d_cycles_per_warp_main_3_semi_join_build_6, d_part__p_name, d_part__p_partkey, part_size);
int64_t* d_cycles_per_warp_main_7_semi_join_probe_6;
auto main_7_semi_join_probe_6_cpw_size = std::ceil((float)partsupp_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_semi_join_probe_6, sizeof(int64_t) * main_7_semi_join_probe_6_cpw_size);
hipMemset(d_cycles_per_warp_main_7_semi_join_probe_6, -1, sizeof(int64_t) * main_7_semi_join_probe_6_cpw_size);
int64_t* d_cycles_per_warp_main_7_join_build_8;
auto main_7_join_build_8_cpw_size = std::ceil((float)partsupp_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_7_join_build_8, sizeof(int64_t) * main_7_join_build_8_cpw_size);
hipMemset(d_cycles_per_warp_main_7_join_build_8, -1, sizeof(int64_t) * main_7_join_build_8_cpw_size);
size_t COUNT8 = partsupp_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_8;
hipMalloc(&d_BUF_IDX_8, sizeof(uint64_t));
hipMemset(d_BUF_IDX_8, 0, sizeof(uint64_t));
uint64_t* d_BUF_8;
hipMalloc(&d_BUF_8, sizeof(uint64_t) * COUNT8 * 1);
auto d_HT_8 = cuco::static_map{ (int)COUNT8*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)partsupp_size/(float)TILE_SIZE), TB>>>(d_BUF_8, d_BUF_IDX_8, d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::insert), d_cycles_per_warp_main_7_join_build_8, d_cycles_per_warp_main_7_semi_join_probe_6, d_partsupp__ps_partkey, d_partsupp__ps_suppkey, partsupp_size);
int64_t* cycles_per_warp_main_7_semi_join_probe_6 = (int64_t*)malloc(sizeof(int64_t) * main_7_semi_join_probe_6_cpw_size);
hipMemcpy(cycles_per_warp_main_7_semi_join_probe_6, d_cycles_per_warp_main_7_semi_join_probe_6, sizeof(int64_t) * main_7_semi_join_probe_6_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_semi_join_probe_6 ";
for (auto i=0ull; i < main_7_semi_join_probe_6_cpw_size; i++) std::cout << cycles_per_warp_main_7_semi_join_probe_6[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_7_join_build_8 = (int64_t*)malloc(sizeof(int64_t) * main_7_join_build_8_cpw_size);
hipMemcpy(cycles_per_warp_main_7_join_build_8, d_cycles_per_warp_main_7_join_build_8, sizeof(int64_t) * main_7_join_build_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_7_join_build_8 ";
for (auto i=0ull; i < main_7_join_build_8_cpw_size; i++) std::cout << cycles_per_warp_main_7_join_build_8[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_5_selection_4;
auto main_5_selection_4_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_selection_4, sizeof(int64_t) * main_5_selection_4_cpw_size);
hipMemset(d_cycles_per_warp_main_5_selection_4, -1, sizeof(int64_t) * main_5_selection_4_cpw_size);
int64_t* d_cycles_per_warp_main_5_join_probe_8;
auto main_5_join_probe_8_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_join_probe_8, sizeof(int64_t) * main_5_join_probe_8_cpw_size);
hipMemset(d_cycles_per_warp_main_5_join_probe_8, -1, sizeof(int64_t) * main_5_join_probe_8_cpw_size);
int64_t* d_cycles_per_warp_main_5_aggregation_9;
auto main_5_aggregation_9_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_5_aggregation_9, sizeof(int64_t) * main_5_aggregation_9_cpw_size);
hipMemset(d_cycles_per_warp_main_5_aggregation_9, -1, sizeof(int64_t) * main_5_aggregation_9_cpw_size);
size_t COUNT9 = 861503;
auto d_HT_9 = cuco::static_map{ (int)861503*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_9;
hipMalloc(&d_SLOT_COUNT_9, sizeof(int));
hipMemset(d_SLOT_COUNT_9, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT9);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT9);
DBI32Type* d_moved_aggr__ps_suppkey;
hipMalloc(&d_moved_aggr__ps_suppkey, sizeof(DBI32Type) * COUNT9);
hipMemset(d_moved_aggr__ps_suppkey, 0, sizeof(DBI32Type) * COUNT9);
DBI32Type* d_moved_aggr_u_1__ps_availqty;
hipMalloc(&d_moved_aggr_u_1__ps_availqty, sizeof(DBI32Type) * COUNT9);
hipMemset(d_moved_aggr_u_1__ps_availqty, 0, sizeof(DBI32Type) * COUNT9);
DBI32Type* d_moved_aggr_u_2__ps_partkey;
hipMalloc(&d_moved_aggr_u_2__ps_partkey, sizeof(DBI32Type) * COUNT9);
hipMemset(d_moved_aggr_u_2__ps_partkey, 0, sizeof(DBI32Type) * COUNT9);
DBI32Type* d_KEY_9lineitem__l_suppkey;
hipMalloc(&d_KEY_9lineitem__l_suppkey, sizeof(DBI32Type) * COUNT9);
hipMemset(d_KEY_9lineitem__l_suppkey, 0, sizeof(DBI32Type) * COUNT9);
DBI32Type* d_KEY_9lineitem__l_partkey;
hipMalloc(&d_KEY_9lineitem__l_partkey, sizeof(DBI32Type) * COUNT9);
hipMemset(d_KEY_9lineitem__l_partkey, 0, sizeof(DBI32Type) * COUNT9);
main_5<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TB>>>(d_BUF_8, d_HT_8.ref(cuco::find), d_HT_9.ref(cuco::insert_and_find), d_KEY_9lineitem__l_partkey, d_KEY_9lineitem__l_suppkey, d_SLOT_COUNT_9, d_aggr0__tmp_attr0, d_cycles_per_warp_main_5_aggregation_9, d_cycles_per_warp_main_5_join_probe_8, d_cycles_per_warp_main_5_selection_4, d_lineitem__l_partkey, d_lineitem__l_quantity, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_moved_aggr__ps_suppkey, d_moved_aggr_u_1__ps_availqty, d_moved_aggr_u_2__ps_partkey, d_partsupp__ps_availqty, d_partsupp__ps_partkey, d_partsupp__ps_suppkey);
int64_t* cycles_per_warp_main_5_selection_4 = (int64_t*)malloc(sizeof(int64_t) * main_5_selection_4_cpw_size);
hipMemcpy(cycles_per_warp_main_5_selection_4, d_cycles_per_warp_main_5_selection_4, sizeof(int64_t) * main_5_selection_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_selection_4 ";
for (auto i=0ull; i < main_5_selection_4_cpw_size; i++) std::cout << cycles_per_warp_main_5_selection_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_join_probe_8 = (int64_t*)malloc(sizeof(int64_t) * main_5_join_probe_8_cpw_size);
hipMemcpy(cycles_per_warp_main_5_join_probe_8, d_cycles_per_warp_main_5_join_probe_8, sizeof(int64_t) * main_5_join_probe_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_join_probe_8 ";
for (auto i=0ull; i < main_5_join_probe_8_cpw_size; i++) std::cout << cycles_per_warp_main_5_join_probe_8[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_5_aggregation_9 = (int64_t*)malloc(sizeof(int64_t) * main_5_aggregation_9_cpw_size);
hipMemcpy(cycles_per_warp_main_5_aggregation_9, d_cycles_per_warp_main_5_aggregation_9, sizeof(int64_t) * main_5_aggregation_9_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_5_aggregation_9 ";
for (auto i=0ull; i < main_5_aggregation_9_cpw_size; i++) std::cout << cycles_per_warp_main_5_aggregation_9[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_join_build_10;
auto main_1_join_build_10_cpw_size = std::ceil((float)nation_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_build_10, sizeof(int64_t) * main_1_join_build_10_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_build_10, -1, sizeof(int64_t) * main_1_join_build_10_cpw_size);
size_t COUNT10 = nation_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_10;
hipMalloc(&d_BUF_IDX_10, sizeof(uint64_t));
hipMemset(d_BUF_IDX_10, 0, sizeof(uint64_t));
uint64_t* d_BUF_10;
hipMalloc(&d_BUF_10, sizeof(uint64_t) * COUNT10 * 1);
auto d_HT_10 = cuco::static_map{ (int)COUNT10*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)nation_size/(float)TILE_SIZE), TB>>>(d_BUF_10, d_BUF_IDX_10, d_HT_10.ref(cuco::insert), d_cycles_per_warp_main_1_join_build_10, d_cycles_per_warp_main_1_selection_0, d_nation__n_name, d_nation__n_nationkey, nation_size);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_build_10 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_build_10_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_build_10, d_cycles_per_warp_main_1_join_build_10, sizeof(int64_t) * main_1_join_build_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_build_10 ";
for (auto i=0ull; i < main_1_join_build_10_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_build_10[i] << " ";
std::cout << std::endl;
COUNT9 = d_HT_9.size();
int64_t* d_cycles_per_warp_main_13_map_12;
auto main_13_map_12_cpw_size = std::ceil((float)COUNT9/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_13_map_12, sizeof(int64_t) * main_13_map_12_cpw_size);
hipMemset(d_cycles_per_warp_main_13_map_12, -1, sizeof(int64_t) * main_13_map_12_cpw_size);
int64_t* d_cycles_per_warp_main_13_selection_14;
auto main_13_selection_14_cpw_size = std::ceil((float)COUNT9/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_13_selection_14, sizeof(int64_t) * main_13_selection_14_cpw_size);
hipMemset(d_cycles_per_warp_main_13_selection_14, -1, sizeof(int64_t) * main_13_selection_14_cpw_size);
int64_t* d_cycles_per_warp_main_13_selection_15;
auto main_13_selection_15_cpw_size = std::ceil((float)COUNT9/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_13_selection_15, sizeof(int64_t) * main_13_selection_15_cpw_size);
hipMemset(d_cycles_per_warp_main_13_selection_15, -1, sizeof(int64_t) * main_13_selection_15_cpw_size);
int64_t* d_cycles_per_warp_main_13_semi_join_build_16;
auto main_13_semi_join_build_16_cpw_size = std::ceil((float)COUNT9/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_13_semi_join_build_16, sizeof(int64_t) * main_13_semi_join_build_16_cpw_size);
hipMemset(d_cycles_per_warp_main_13_semi_join_build_16, -1, sizeof(int64_t) * main_13_semi_join_build_16_cpw_size);
size_t COUNT16 = COUNT9;
// Insert hash table control;
auto d_HT_16 = cuco::static_map{ (int)COUNT16*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_13<<<std::ceil((float)COUNT9/(float)TILE_SIZE), TB>>>(COUNT9, d_HT_16.ref(cuco::insert), d_aggr0__tmp_attr0, d_cycles_per_warp_main_13_map_12, d_cycles_per_warp_main_13_selection_14, d_cycles_per_warp_main_13_selection_15, d_cycles_per_warp_main_13_semi_join_build_16, d_moved_aggr__ps_suppkey, d_moved_aggr_u_1__ps_availqty, d_moved_aggr_u_2__ps_partkey);
int64_t* d_cycles_per_warp_main_11_join_probe_10;
auto main_11_join_probe_10_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_join_probe_10, sizeof(int64_t) * main_11_join_probe_10_cpw_size);
hipMemset(d_cycles_per_warp_main_11_join_probe_10, -1, sizeof(int64_t) * main_11_join_probe_10_cpw_size);
int64_t* d_cycles_per_warp_main_11_semi_join_probe_16;
auto main_11_semi_join_probe_16_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_semi_join_probe_16, sizeof(int64_t) * main_11_semi_join_probe_16_cpw_size);
hipMemset(d_cycles_per_warp_main_11_semi_join_probe_16, -1, sizeof(int64_t) * main_11_semi_join_probe_16_cpw_size);
int64_t* d_cycles_per_warp_main_11_materialize_17;
auto main_11_materialize_17_cpw_size = std::ceil((float)supplier_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_materialize_17, sizeof(int64_t) * main_11_materialize_17_cpw_size);
hipMemset(d_cycles_per_warp_main_11_materialize_17, -1, sizeof(int64_t) * main_11_materialize_17_cpw_size);
size_t COUNT17 = supplier_size;
//Materialize buffers
uint64_t* d_MAT_IDX17;
hipMalloc(&d_MAT_IDX17, sizeof(uint64_t));
hipMemset(d_MAT_IDX17, 0, sizeof(uint64_t));
auto MAT17supplier__s_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT17);
DBI16Type* d_MAT17supplier__s_name_encoded;
hipMalloc(&d_MAT17supplier__s_name_encoded, sizeof(DBI16Type) * COUNT17);
auto MAT17supplier__s_address_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT17);
DBI16Type* d_MAT17supplier__s_address_encoded;
hipMalloc(&d_MAT17supplier__s_address_encoded, sizeof(DBI16Type) * COUNT17);
main_11<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TB>>>(d_BUF_10, d_HT_10.ref(cuco::find), d_HT_16.ref(cuco::find), d_MAT17supplier__s_address_encoded, d_MAT17supplier__s_name_encoded, d_MAT_IDX17, d_cycles_per_warp_main_11_join_probe_10, d_cycles_per_warp_main_11_materialize_17, d_cycles_per_warp_main_11_semi_join_probe_16, d_supplier__s_address_encoded, d_supplier__s_name_encoded, d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
uint64_t MATCOUNT_17 = 0;
hipMemcpy(&MATCOUNT_17, d_MAT_IDX17, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT17supplier__s_name_encoded, d_MAT17supplier__s_name_encoded, sizeof(DBI16Type) * COUNT17, hipMemcpyDeviceToHost);
hipMemcpy(MAT17supplier__s_address_encoded, d_MAT17supplier__s_address_encoded, sizeof(DBI16Type) * COUNT17, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_11_join_probe_10 = (int64_t*)malloc(sizeof(int64_t) * main_11_join_probe_10_cpw_size);
hipMemcpy(cycles_per_warp_main_11_join_probe_10, d_cycles_per_warp_main_11_join_probe_10, sizeof(int64_t) * main_11_join_probe_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_join_probe_10 ";
for (auto i=0ull; i < main_11_join_probe_10_cpw_size; i++) std::cout << cycles_per_warp_main_11_join_probe_10[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_11_semi_join_probe_16 = (int64_t*)malloc(sizeof(int64_t) * main_11_semi_join_probe_16_cpw_size);
hipMemcpy(cycles_per_warp_main_11_semi_join_probe_16, d_cycles_per_warp_main_11_semi_join_probe_16, sizeof(int64_t) * main_11_semi_join_probe_16_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_semi_join_probe_16 ";
for (auto i=0ull; i < main_11_semi_join_probe_16_cpw_size; i++) std::cout << cycles_per_warp_main_11_semi_join_probe_16[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_11_materialize_17 = (int64_t*)malloc(sizeof(int64_t) * main_11_materialize_17_cpw_size);
hipMemcpy(cycles_per_warp_main_11_materialize_17, d_cycles_per_warp_main_11_materialize_17, sizeof(int64_t) * main_11_materialize_17_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_materialize_17 ";
for (auto i=0ull; i < main_11_materialize_17_cpw_size; i++) std::cout << cycles_per_warp_main_11_materialize_17[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_8);
hipFree(d_BUF_IDX_8);
hipFree(d_KEY_9lineitem__l_partkey);
hipFree(d_KEY_9lineitem__l_suppkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_moved_aggr__ps_suppkey);
hipFree(d_moved_aggr_u_1__ps_availqty);
hipFree(d_moved_aggr_u_2__ps_partkey);
hipFree(d_BUF_10);
hipFree(d_BUF_IDX_10);
hipFree(d_MAT17supplier__s_address_encoded);
hipFree(d_MAT17supplier__s_name_encoded);
hipFree(d_MAT_IDX17);
free(MAT17supplier__s_address_encoded);
free(MAT17supplier__s_name_encoded);
}