#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_6(uint64_t* COUNT5, size_t orders_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT5, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_6(uint64_t* BUF_5, uint64_t* BUF_IDX_5, HASHTABLE_INSERT HT_5, int64_t* cycles_per_warp_main_6_join_build_5, DBI32Type* orders__o_orderkey, size_t orders_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_5[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_5[ITEM] = 0;
KEY_5[ITEM] |= reg_orders__o_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_5 = atomicAdd((int*)BUF_IDX_5, 1);
HT_5.insert(cuco::pair{KEY_5[ITEM], buf_idx_5});
BUF_5[(buf_idx_5) * 1 + 0] = ITEM*TB + tid;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_6_join_build_5[blockIdx.x] = cycles_per_warp;}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_1(uint64_t* BUF_5, HASHTABLE_PROBE HT_5, HASHTABLE_INSERT HT_9, DBDateType* lineitem__l_commitdate, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_receiptdate, DBDateType* lineitem__l_shipdate, DBStringType* lineitem__l_shipmode, DBI16Type* lineitem__l_shipmode_encoded, size_t lineitem_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_lineitem__l_receiptdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_receiptdate[ITEM] = lineitem__l_receiptdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_receiptdate[ITEM], 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_receiptdate[ITEM], 9131, Predicate::lt);
}
DBDateType reg_lineitem__l_shipdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipdate[ITEM] = lineitem__l_shipdate[ITEM*TB + tid];
}
DBDateType reg_lineitem__l_commitdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_commitdate[ITEM] = lineitem__l_commitdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_shipdate[ITEM], reg_lineitem__l_commitdate[ITEM], Predicate::lt);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_commitdate[ITEM], reg_lineitem__l_receiptdate[ITEM], Predicate::lt);
}
DBStringType reg_lineitem__l_shipmode[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipmode[ITEM] = lineitem__l_shipmode[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_lineitem__l_shipmode[ITEM], "MAIL", Predicate::eq)) || (evaluatePredicate(reg_lineitem__l_shipmode[ITEM], "SHIP", Predicate::eq));
}
uint64_t KEY_5[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_5[ITEM] = 0;
KEY_5[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
int64_t slot_second5[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_5 = HT_5.find(KEY_5[ITEM]);
if (SLOT_5 == HT_5.end()) {selection_flags[ITEM] = 0; continue;}
slot_second5[ITEM] = SLOT_5->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_9[ITEMS_PER_THREAD];
DBI16Type reg_lineitem__l_shipmode_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipmode_encoded[ITEM] = lineitem__l_shipmode_encoded[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_9[ITEM] = 0;
KEY_9[ITEM] |= reg_lineitem__l_shipmode_encoded[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_9.insert(cuco::pair{KEY_9[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_1(uint64_t* BUF_5, HASHTABLE_PROBE HT_5, HASHTABLE_FIND HT_9, DBI16Type* KEY_9lineitem__l_shipmode_encoded, DBI32Type* aggr0__tmp_attr0, DBI32Type* aggr0__tmp_attr2, int64_t* cycles_per_warp_main_1_aggregation_9, int64_t* cycles_per_warp_main_1_join_probe_5, int64_t* cycles_per_warp_main_1_map_7, int64_t* cycles_per_warp_main_1_map_8, int64_t* cycles_per_warp_main_1_selection_0, int64_t* cycles_per_warp_main_1_selection_2, int64_t* cycles_per_warp_main_1_selection_3, int64_t* cycles_per_warp_main_1_selection_4, DBDateType* lineitem__l_commitdate, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_receiptdate, DBDateType* lineitem__l_shipdate, DBStringType* lineitem__l_shipmode, DBI16Type* lineitem__l_shipmode_encoded, size_t lineitem_size, DBStringType* orders__o_orderpriority) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
DBDateType reg_lineitem__l_receiptdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_receiptdate[ITEM] = lineitem__l_receiptdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_receiptdate[ITEM], 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_receiptdate[ITEM], 9131, Predicate::lt);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_0[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
DBDateType reg_lineitem__l_shipdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipdate[ITEM] = lineitem__l_shipdate[ITEM*TB + tid];
}
DBDateType reg_lineitem__l_commitdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_commitdate[ITEM] = lineitem__l_commitdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_shipdate[ITEM], reg_lineitem__l_commitdate[ITEM], Predicate::lt);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_2[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_commitdate[ITEM], reg_lineitem__l_receiptdate[ITEM], Predicate::lt);
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_3[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
DBStringType reg_lineitem__l_shipmode[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipmode[ITEM] = lineitem__l_shipmode[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(reg_lineitem__l_shipmode[ITEM], "MAIL", Predicate::eq)) || (evaluatePredicate(reg_lineitem__l_shipmode[ITEM], "SHIP", Predicate::eq));
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_selection_4[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_5[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_5[ITEM] = 0;
KEY_5[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
int64_t slot_second5[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_5 = HT_5.find(KEY_5[ITEM]);
if (SLOT_5 == HT_5.end()) {selection_flags[ITEM] = 0; continue;}
slot_second5[ITEM] = SLOT_5->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_join_probe_5[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_map_7[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_map_8[blockIdx.x] = cycles_per_warp;}
if (threadIdx.x == 0) start = clock64();
uint64_t KEY_9[ITEMS_PER_THREAD];
DBI16Type reg_lineitem__l_shipmode_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipmode_encoded[ITEM] = lineitem__l_shipmode_encoded[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_9[ITEM] = 0;
KEY_9[ITEM] |= reg_lineitem__l_shipmode_encoded[ITEM];
}
//Aggregate in hashtable
DBStringType reg_orders__o_orderpriority[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_orderpriority[ITEM] = orders__o_orderpriority[BUF_5[slot_second5[ITEM] * 1 + 0]];
}
DBI32Type reg_map0__tmp_attr3[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr3[ITEM] = (((evaluatePredicate(reg_orders__o_orderpriority[ITEM], "1-URGENT", Predicate::neq)) && (evaluatePredicate(reg_orders__o_orderpriority[ITEM], "2-HIGH", Predicate::neq))));
}
DBI32Type reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (((evaluatePredicate(reg_orders__o_orderpriority[ITEM], "1-URGENT", Predicate::eq)) || (evaluatePredicate(reg_orders__o_orderpriority[ITEM], "2-HIGH", Predicate::eq))));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_9 = HT_9.find(KEY_9[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr2[buf_idx_9], reg_map0__tmp_attr3[ITEM]);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_9], reg_map0__tmp_attr1[ITEM]);
KEY_9lineitem__l_shipmode_encoded[buf_idx_9] = reg_lineitem__l_shipmode_encoded[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_1_aggregation_9[blockIdx.x] = cycles_per_warp;}
}
__global__ void count_11(uint64_t* COUNT10, size_t COUNT9) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT10, 1);
}
}
__global__ void main_11(size_t COUNT9, DBI32Type* MAT10aggr0__tmp_attr0, DBI32Type* MAT10aggr0__tmp_attr2, DBI16Type* MAT10lineitem__l_shipmode_encoded, uint64_t* MAT_IDX10, DBI32Type* aggr0__tmp_attr0, DBI32Type* aggr0__tmp_attr2, int64_t* cycles_per_warp_main_11_materialize_10, DBI16Type* lineitem__l_shipmode_encoded) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
if (threadIdx.x == 0) start = clock64();
//Materialize buffers
DBI16Type reg_lineitem__l_shipmode_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
reg_lineitem__l_shipmode_encoded[ITEM] = lineitem__l_shipmode_encoded[ITEM*TB + tid];
}
DBI32Type reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
DBI32Type reg_aggr0__tmp_attr2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
reg_aggr0__tmp_attr2[ITEM] = aggr0__tmp_attr2[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT9); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx10 = atomicAdd((int*)MAT_IDX10, 1);
MAT10lineitem__l_shipmode_encoded[mat_idx10] = reg_lineitem__l_shipmode_encoded[ITEM];
MAT10aggr0__tmp_attr0[mat_idx10] = reg_aggr0__tmp_attr0[ITEM];
MAT10aggr0__tmp_attr2[mat_idx10] = reg_aggr0__tmp_attr2[ITEM];
}
if (threadIdx.x == 0) {            stop = clock64();            cycles_per_warp = (stop - start);            cycles_per_warp_main_11_materialize_10[blockIdx.x] = cycles_per_warp;}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
//Materialize count
uint64_t* d_COUNT5;
hipMalloc(&d_COUNT5, sizeof(uint64_t));
hipMemset(d_COUNT5, 0, sizeof(uint64_t));
count_6<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT5, orders_size);
uint64_t COUNT5;
hipMemcpy(&COUNT5, d_COUNT5, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_6_join_build_5;
auto main_6_join_build_5_cpw_size = std::ceil((float)orders_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_6_join_build_5, sizeof(int64_t) * main_6_join_build_5_cpw_size);
hipMemset(d_cycles_per_warp_main_6_join_build_5, -1, sizeof(int64_t) * main_6_join_build_5_cpw_size);
// Insert hash table control;
uint64_t* d_BUF_IDX_5;
hipMalloc(&d_BUF_IDX_5, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5, 0, sizeof(uint64_t));
uint64_t* d_BUF_5;
hipMalloc(&d_BUF_5, sizeof(uint64_t) * COUNT5 * 1);
auto d_HT_5 = cuco::static_map{ (int)COUNT5*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_6<<<std::ceil((float)orders_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_5, d_BUF_IDX_5, d_HT_5.ref(cuco::insert), d_cycles_per_warp_main_6_join_build_5, d_orders__o_orderkey, orders_size);
int64_t* cycles_per_warp_main_6_join_build_5 = (int64_t*)malloc(sizeof(int64_t) * main_6_join_build_5_cpw_size);
hipMemcpy(cycles_per_warp_main_6_join_build_5, d_cycles_per_warp_main_6_join_build_5, sizeof(int64_t) * main_6_join_build_5_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_6_join_build_5 ";
for (auto i=0ull; i < main_6_join_build_5_cpw_size; i++) std::cout << cycles_per_warp_main_6_join_build_5[i] << " ";
std::cout << std::endl;
int64_t* d_cycles_per_warp_main_1_selection_0;
auto main_1_selection_0_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_0, -1, sizeof(int64_t) * main_1_selection_0_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_2;
auto main_1_selection_2_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_2, -1, sizeof(int64_t) * main_1_selection_2_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_3;
auto main_1_selection_3_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_3, -1, sizeof(int64_t) * main_1_selection_3_cpw_size);
int64_t* d_cycles_per_warp_main_1_selection_4;
auto main_1_selection_4_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_selection_4, sizeof(int64_t) * main_1_selection_4_cpw_size);
hipMemset(d_cycles_per_warp_main_1_selection_4, -1, sizeof(int64_t) * main_1_selection_4_cpw_size);
int64_t* d_cycles_per_warp_main_1_join_probe_5;
auto main_1_join_probe_5_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_join_probe_5, sizeof(int64_t) * main_1_join_probe_5_cpw_size);
hipMemset(d_cycles_per_warp_main_1_join_probe_5, -1, sizeof(int64_t) * main_1_join_probe_5_cpw_size);
int64_t* d_cycles_per_warp_main_1_map_7;
auto main_1_map_7_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_map_7, sizeof(int64_t) * main_1_map_7_cpw_size);
hipMemset(d_cycles_per_warp_main_1_map_7, -1, sizeof(int64_t) * main_1_map_7_cpw_size);
int64_t* d_cycles_per_warp_main_1_map_8;
auto main_1_map_8_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_map_8, sizeof(int64_t) * main_1_map_8_cpw_size);
hipMemset(d_cycles_per_warp_main_1_map_8, -1, sizeof(int64_t) * main_1_map_8_cpw_size);
//Create aggregation hash table
auto d_HT_9 = cuco::static_map{ (int)17582*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_1<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_5, d_HT_5.ref(cuco::find), d_HT_9.ref(cuco::insert), d_lineitem__l_commitdate, d_lineitem__l_orderkey, d_lineitem__l_receiptdate, d_lineitem__l_shipdate, d_lineitem__l_shipmode, d_lineitem__l_shipmode_encoded, lineitem_size);
size_t COUNT9 = d_HT_9.size();
thrust::device_vector<int64_t> keys_9(COUNT9), vals_9(COUNT9);
d_HT_9.retrieve_all(keys_9.begin(), vals_9.begin());
d_HT_9.clear();
int64_t* raw_keys9 = thrust::raw_pointer_cast(keys_9.data());
insertKeys<<<std::ceil((float)COUNT9/128.), 128>>>(raw_keys9, d_HT_9.ref(cuco::insert), COUNT9);
int64_t* d_cycles_per_warp_main_1_aggregation_9;
auto main_1_aggregation_9_cpw_size = std::ceil((float)lineitem_size/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_1_aggregation_9, sizeof(int64_t) * main_1_aggregation_9_cpw_size);
hipMemset(d_cycles_per_warp_main_1_aggregation_9, -1, sizeof(int64_t) * main_1_aggregation_9_cpw_size);
//Aggregate in hashtable
DBI32Type* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBI32Type) * COUNT9);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBI32Type) * COUNT9);
DBI32Type* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBI32Type) * COUNT9);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBI32Type) * COUNT9);
DBI16Type* d_KEY_9lineitem__l_shipmode_encoded;
hipMalloc(&d_KEY_9lineitem__l_shipmode_encoded, sizeof(DBI16Type) * COUNT9);
hipMemset(d_KEY_9lineitem__l_shipmode_encoded, 0, sizeof(DBI16Type) * COUNT9);
main_1<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_5, d_HT_5.ref(cuco::find), d_HT_9.ref(cuco::find), d_KEY_9lineitem__l_shipmode_encoded, d_aggr0__tmp_attr0, d_aggr0__tmp_attr2, d_cycles_per_warp_main_1_aggregation_9, d_cycles_per_warp_main_1_join_probe_5, d_cycles_per_warp_main_1_map_7, d_cycles_per_warp_main_1_map_8, d_cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_3, d_cycles_per_warp_main_1_selection_4, d_lineitem__l_commitdate, d_lineitem__l_orderkey, d_lineitem__l_receiptdate, d_lineitem__l_shipdate, d_lineitem__l_shipmode, d_lineitem__l_shipmode_encoded, lineitem_size, d_orders__o_orderpriority);
int64_t* cycles_per_warp_main_1_selection_0 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_0_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_0, d_cycles_per_warp_main_1_selection_0, sizeof(int64_t) * main_1_selection_0_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_0 ";
for (auto i=0ull; i < main_1_selection_0_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_0[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_2 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_2_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_2, d_cycles_per_warp_main_1_selection_2, sizeof(int64_t) * main_1_selection_2_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_2 ";
for (auto i=0ull; i < main_1_selection_2_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_2[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_3 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_3_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_3, d_cycles_per_warp_main_1_selection_3, sizeof(int64_t) * main_1_selection_3_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_3 ";
for (auto i=0ull; i < main_1_selection_3_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_3[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_selection_4 = (int64_t*)malloc(sizeof(int64_t) * main_1_selection_4_cpw_size);
hipMemcpy(cycles_per_warp_main_1_selection_4, d_cycles_per_warp_main_1_selection_4, sizeof(int64_t) * main_1_selection_4_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_selection_4 ";
for (auto i=0ull; i < main_1_selection_4_cpw_size; i++) std::cout << cycles_per_warp_main_1_selection_4[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_join_probe_5 = (int64_t*)malloc(sizeof(int64_t) * main_1_join_probe_5_cpw_size);
hipMemcpy(cycles_per_warp_main_1_join_probe_5, d_cycles_per_warp_main_1_join_probe_5, sizeof(int64_t) * main_1_join_probe_5_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_join_probe_5 ";
for (auto i=0ull; i < main_1_join_probe_5_cpw_size; i++) std::cout << cycles_per_warp_main_1_join_probe_5[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_map_7 = (int64_t*)malloc(sizeof(int64_t) * main_1_map_7_cpw_size);
hipMemcpy(cycles_per_warp_main_1_map_7, d_cycles_per_warp_main_1_map_7, sizeof(int64_t) * main_1_map_7_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_map_7 ";
for (auto i=0ull; i < main_1_map_7_cpw_size; i++) std::cout << cycles_per_warp_main_1_map_7[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_map_8 = (int64_t*)malloc(sizeof(int64_t) * main_1_map_8_cpw_size);
hipMemcpy(cycles_per_warp_main_1_map_8, d_cycles_per_warp_main_1_map_8, sizeof(int64_t) * main_1_map_8_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_map_8 ";
for (auto i=0ull; i < main_1_map_8_cpw_size; i++) std::cout << cycles_per_warp_main_1_map_8[i] << " ";
std::cout << std::endl;
int64_t* cycles_per_warp_main_1_aggregation_9 = (int64_t*)malloc(sizeof(int64_t) * main_1_aggregation_9_cpw_size);
hipMemcpy(cycles_per_warp_main_1_aggregation_9, d_cycles_per_warp_main_1_aggregation_9, sizeof(int64_t) * main_1_aggregation_9_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_1_aggregation_9 ";
for (auto i=0ull; i < main_1_aggregation_9_cpw_size; i++) std::cout << cycles_per_warp_main_1_aggregation_9[i] << " ";
std::cout << std::endl;
//Materialize count
uint64_t* d_COUNT10;
hipMalloc(&d_COUNT10, sizeof(uint64_t));
hipMemset(d_COUNT10, 0, sizeof(uint64_t));
count_11<<<std::ceil((float)COUNT9/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT10, COUNT9);
uint64_t COUNT10;
hipMemcpy(&COUNT10, d_COUNT10, sizeof(uint64_t), hipMemcpyDeviceToHost);
int64_t* d_cycles_per_warp_main_11_materialize_10;
auto main_11_materialize_10_cpw_size = std::ceil((float)COUNT9/(float)TILE_SIZE);
hipMalloc(&d_cycles_per_warp_main_11_materialize_10, sizeof(int64_t) * main_11_materialize_10_cpw_size);
hipMemset(d_cycles_per_warp_main_11_materialize_10, -1, sizeof(int64_t) * main_11_materialize_10_cpw_size);
//Materialize buffers
uint64_t* d_MAT_IDX10;
hipMalloc(&d_MAT_IDX10, sizeof(uint64_t));
hipMemset(d_MAT_IDX10, 0, sizeof(uint64_t));
auto MAT10lineitem__l_shipmode_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT10);
DBI16Type* d_MAT10lineitem__l_shipmode_encoded;
hipMalloc(&d_MAT10lineitem__l_shipmode_encoded, sizeof(DBI16Type) * COUNT10);
auto MAT10aggr0__tmp_attr0 = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT10);
DBI32Type* d_MAT10aggr0__tmp_attr0;
hipMalloc(&d_MAT10aggr0__tmp_attr0, sizeof(DBI32Type) * COUNT10);
auto MAT10aggr0__tmp_attr2 = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT10);
DBI32Type* d_MAT10aggr0__tmp_attr2;
hipMalloc(&d_MAT10aggr0__tmp_attr2, sizeof(DBI32Type) * COUNT10);
main_11<<<std::ceil((float)COUNT9/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT9, d_MAT10aggr0__tmp_attr0, d_MAT10aggr0__tmp_attr2, d_MAT10lineitem__l_shipmode_encoded, d_MAT_IDX10, d_aggr0__tmp_attr0, d_aggr0__tmp_attr2, d_cycles_per_warp_main_11_materialize_10, d_KEY_9lineitem__l_shipmode_encoded);
hipMemcpy(MAT10lineitem__l_shipmode_encoded, d_MAT10lineitem__l_shipmode_encoded, sizeof(DBI16Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10aggr0__tmp_attr0, d_MAT10aggr0__tmp_attr0, sizeof(DBI32Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10aggr0__tmp_attr2, d_MAT10aggr0__tmp_attr2, sizeof(DBI32Type) * COUNT10, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
int64_t* cycles_per_warp_main_11_materialize_10 = (int64_t*)malloc(sizeof(int64_t) * main_11_materialize_10_cpw_size);
hipMemcpy(cycles_per_warp_main_11_materialize_10, d_cycles_per_warp_main_11_materialize_10, sizeof(int64_t) * main_11_materialize_10_cpw_size, hipMemcpyDeviceToHost);
std::cout << "main_11_materialize_10 ";
for (auto i=0ull; i < main_11_materialize_10_cpw_size; i++) std::cout << cycles_per_warp_main_11_materialize_10[i] << " ";
std::cout << std::endl;
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
      size_t aux_mem = usedGpuMem() - used_mem;
      std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_5);
hipFree(d_BUF_IDX_5);
hipFree(d_COUNT5);
hipFree(d_KEY_9lineitem__l_shipmode_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_COUNT10);
hipFree(d_MAT10aggr0__tmp_attr0);
hipFree(d_MAT10aggr0__tmp_attr2);
hipFree(d_MAT10lineitem__l_shipmode_encoded);
hipFree(d_MAT_IDX10);
free(MAT10aggr0__tmp_attr0);
free(MAT10aggr0__tmp_attr2);
free(MAT10lineitem__l_shipmode_encoded);
}