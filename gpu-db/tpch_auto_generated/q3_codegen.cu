#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_63411b08e9e0(uint64_t* COUNT63411b17f960, DBStringType* customer__c_mktsegment, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_mktsegment = customer__c_mktsegment[tid];
if (!(evaluatePredicate(reg_customer__c_mktsegment, "BUILDING", Predicate::eq))) return;
//Materialize count
atomicAdd((int*)COUNT63411b17f960, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_63411b08e9e0(uint64_t* BUF_63411b17f960, uint64_t* BUF_IDX_63411b17f960, HASHTABLE_INSERT HT_63411b17f960, DBI32Type* customer__c_custkey, DBStringType* customer__c_mktsegment, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_mktsegment = customer__c_mktsegment[tid];
if (!(evaluatePredicate(reg_customer__c_mktsegment, "BUILDING", Predicate::eq))) return;
uint64_t KEY_63411b17f960 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];
KEY_63411b17f960 <<= 32;
KEY_63411b17f960 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_63411b17f960 = atomicAdd((int*)BUF_IDX_63411b17f960, 1);
HT_63411b17f960.insert(cuco::pair{KEY_63411b17f960, buf_idx_63411b17f960});
BUF_63411b17f960[buf_idx_63411b17f960 * 1 + 0] = tid;
}
template<typename HASHTABLE_FIND>
__global__ void count_63411b111520(uint64_t* BUF_63411b17f960, uint64_t* COUNT63411b17fc20, HASHTABLE_FIND HT_63411b17f960, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 9204, Predicate::lt))) return;
uint64_t KEY_63411b17f960 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];
KEY_63411b17f960 <<= 32;
KEY_63411b17f960 |= reg_orders__o_custkey;
//Probe Hash table
auto SLOT_63411b17f960 = HT_63411b17f960.find(KEY_63411b17f960);
if (SLOT_63411b17f960 == HT_63411b17f960.end()) return;
//Materialize count
atomicAdd((int*)COUNT63411b17fc20, 1);
}
template<typename HASHTABLE_FIND, typename HASHTABLE_INSERT>
__global__ void main_63411b111520(uint64_t* BUF_63411b17f960, uint64_t* BUF_63411b17fc20, uint64_t* BUF_IDX_63411b17fc20, HASHTABLE_FIND HT_63411b17f960, HASHTABLE_INSERT HT_63411b17fc20, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 9204, Predicate::lt))) return;
uint64_t KEY_63411b17f960 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];
KEY_63411b17f960 <<= 32;
KEY_63411b17f960 |= reg_orders__o_custkey;
//Probe Hash table
auto SLOT_63411b17f960 = HT_63411b17f960.find(KEY_63411b17f960);
if (SLOT_63411b17f960 == HT_63411b17f960.end()) return;
uint64_t KEY_63411b17fc20 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];
KEY_63411b17fc20 <<= 32;
KEY_63411b17fc20 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_63411b17fc20 = atomicAdd((int*)BUF_IDX_63411b17fc20, 1);
HT_63411b17fc20.insert(cuco::pair{KEY_63411b17fc20, buf_idx_63411b17fc20});
BUF_63411b17fc20[buf_idx_63411b17fc20 * 2 + 0] = BUF_63411b17f960[SLOT_63411b17f960->second * 1 + 0];
BUF_63411b17fc20[buf_idx_63411b17fc20 * 2 + 1] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_FIND>
__global__ void count_63411b18bef0(uint64_t* BUF_63411b17fc20, HASHTABLE_INSERT HT_63411b13d830, HASHTABLE_FIND HT_63411b17fc20, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9204, Predicate::gt))) return;
uint64_t KEY_63411b17fc20 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];
KEY_63411b17fc20 <<= 32;
KEY_63411b17fc20 |= reg_lineitem__l_orderkey;
//Probe Hash table
auto SLOT_63411b17fc20 = HT_63411b17fc20.find(KEY_63411b17fc20);
if (SLOT_63411b17fc20 == HT_63411b17fc20.end()) return;
uint64_t KEY_63411b13d830 = 0;
KEY_63411b13d830 <<= 32;
KEY_63411b13d830 |= reg_lineitem__l_orderkey;
//Create aggregation hash table
HT_63411b13d830.insert(cuco::pair{KEY_63411b13d830, 1});
}
template<typename HASHTABLE_FIND>
__global__ void main_63411b18bef0(uint64_t* BUF_63411b17fc20, HASHTABLE_FIND HT_63411b13d830, HASHTABLE_FIND HT_63411b17fc20, DBI32Type* KEY_63411b13d830lineitem__l_orderkey, DBDecimalType* aggr0__tmp_attr0, DBDateType* aggr__o_orderdate, DBI32Type* aggr__o_shippriority, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, size_t lineitem_size, DBDateType* orders__o_orderdate, DBI32Type* orders__o_shippriority) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9204, Predicate::gt))) return;
uint64_t KEY_63411b17fc20 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];
KEY_63411b17fc20 <<= 32;
KEY_63411b17fc20 |= reg_lineitem__l_orderkey;
//Probe Hash table
auto SLOT_63411b17fc20 = HT_63411b17fc20.find(KEY_63411b17fc20);
if (SLOT_63411b17fc20 == HT_63411b17fc20.end()) return;
uint64_t KEY_63411b13d830 = 0;
KEY_63411b13d830 <<= 32;
KEY_63411b13d830 |= reg_lineitem__l_orderkey;
//Aggregate in hashtable
auto buf_idx_63411b13d830 = HT_63411b13d830.find(KEY_63411b13d830)->second;
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_63411b13d830], reg_lineitem__l_extendedprice);
auto reg_orders__o_shippriority = orders__o_shippriority[BUF_63411b17fc20[SLOT_63411b17fc20->second * 2 + 0]];
aggregate_any(&aggr__o_shippriority[buf_idx_63411b13d830], reg_orders__o_shippriority);
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_63411b17fc20[SLOT_63411b17fc20->second * 2 + 0]];
aggregate_any(&aggr__o_orderdate[buf_idx_63411b13d830], reg_orders__o_orderdate);
KEY_63411b13d830lineitem__l_orderkey[buf_idx_63411b13d830] = reg_lineitem__l_orderkey;
}
__global__ void count_63411b196f70(uint64_t* COUNT63411b11e300, size_t COUNT63411b13d830) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT63411b13d830) return;
//Materialize count
atomicAdd((int*)COUNT63411b11e300, 1);
}
__global__ void main_63411b196f70(size_t COUNT63411b13d830, DBDecimalType* MAT63411b11e300aggr0__tmp_attr0, DBDateType* MAT63411b11e300aggr__o_orderdate, DBI32Type* MAT63411b11e300aggr__o_shippriority, DBI32Type* MAT63411b11e300lineitem__l_orderkey, DBDecimalType* aggr0__tmp_attr0, DBDateType* aggr__o_orderdate, DBI32Type* aggr__o_shippriority, DBI32Type* lineitem__l_orderkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT63411b13d830) return;
//Materialize buffers
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];
MAT63411b11e300lineitem__l_orderkey[tid] = reg_lineitem__l_orderkey;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT63411b11e300aggr0__tmp_attr0[tid] = reg_aggr0__tmp_attr0;
auto reg_aggr__o_orderdate = aggr__o_orderdate[tid];
MAT63411b11e300aggr__o_orderdate[tid] = reg_aggr__o_orderdate;
auto reg_aggr__o_shippriority = aggr__o_shippriority[tid];
MAT63411b11e300aggr__o_shippriority[tid] = reg_aggr__o_shippriority;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT63411b17f960;
hipMalloc(&d_COUNT63411b17f960, sizeof(uint64_t));
hipMemset(d_COUNT63411b17f960, 0, sizeof(uint64_t));
count_63411b08e9e0<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT63411b17f960, d_customer__c_mktsegment, customer_size);
uint64_t COUNT63411b17f960;
hipMemcpy(&COUNT63411b17f960, d_COUNT63411b17f960, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT63411b17f960);
// Insert hash table control;
uint64_t* d_BUF_IDX_63411b17f960;
hipMalloc(&d_BUF_IDX_63411b17f960, sizeof(uint64_t));
hipMemset(d_BUF_IDX_63411b17f960, 0, sizeof(uint64_t));
uint64_t* d_BUF_63411b17f960;
hipMalloc(&d_BUF_63411b17f960, sizeof(uint64_t) * COUNT63411b17f960 * 1);
auto d_HT_63411b17f960 = cuco::static_map{ (int)COUNT63411b17f960*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_63411b08e9e0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_63411b17f960, d_BUF_IDX_63411b17f960, d_HT_63411b17f960.ref(cuco::insert), d_customer__c_custkey, d_customer__c_mktsegment, customer_size);
hipFree(d_BUF_IDX_63411b17f960);
//Materialize count
uint64_t* d_COUNT63411b17fc20;
hipMalloc(&d_COUNT63411b17fc20, sizeof(uint64_t));
hipMemset(d_COUNT63411b17fc20, 0, sizeof(uint64_t));
count_63411b111520<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_63411b17f960, d_COUNT63411b17fc20, d_HT_63411b17f960.ref(cuco::find), d_orders__o_custkey, d_orders__o_orderdate, orders_size);
uint64_t COUNT63411b17fc20;
hipMemcpy(&COUNT63411b17fc20, d_COUNT63411b17fc20, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT63411b17fc20);
// Insert hash table control;
uint64_t* d_BUF_IDX_63411b17fc20;
hipMalloc(&d_BUF_IDX_63411b17fc20, sizeof(uint64_t));
hipMemset(d_BUF_IDX_63411b17fc20, 0, sizeof(uint64_t));
uint64_t* d_BUF_63411b17fc20;
hipMalloc(&d_BUF_63411b17fc20, sizeof(uint64_t) * COUNT63411b17fc20 * 2);
auto d_HT_63411b17fc20 = cuco::static_map{ (int)COUNT63411b17fc20*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_63411b111520<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_63411b17f960, d_BUF_63411b17fc20, d_BUF_IDX_63411b17fc20, d_HT_63411b17f960.ref(cuco::find), d_HT_63411b17fc20.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
hipFree(d_BUF_IDX_63411b17fc20);
//Create aggregation hash table
auto d_HT_63411b13d830 = cuco::static_map{ (int)355555*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_63411b18bef0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_63411b17fc20, d_HT_63411b13d830.ref(cuco::insert), d_HT_63411b17fc20.ref(cuco::find), d_lineitem__l_orderkey, d_lineitem__l_shipdate, lineitem_size);
size_t COUNT63411b13d830 = d_HT_63411b13d830.size();
thrust::device_vector<int64_t> keys_63411b13d830(COUNT63411b13d830), vals_63411b13d830(COUNT63411b13d830);
d_HT_63411b13d830.retrieve_all(keys_63411b13d830.begin(), vals_63411b13d830.begin());
thrust::host_vector<int64_t> h_keys_63411b13d830(COUNT63411b13d830);
thrust::copy(keys_63411b13d830.begin(), keys_63411b13d830.end(), h_keys_63411b13d830.begin());
thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_63411b13d830(COUNT63411b13d830);
for (int i=0; i < COUNT63411b13d830; i++)
{actual_dict_63411b13d830[i] = cuco::make_pair(h_keys_63411b13d830[i], i);}
d_HT_63411b13d830.clear();
d_HT_63411b13d830.insert(actual_dict_63411b13d830.begin(), actual_dict_63411b13d830.end());
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT63411b13d830);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT63411b13d830);
DBI32Type* d_aggr__o_shippriority;
hipMalloc(&d_aggr__o_shippriority, sizeof(DBI32Type) * COUNT63411b13d830);
hipMemset(d_aggr__o_shippriority, 0, sizeof(DBI32Type) * COUNT63411b13d830);
DBDateType* d_aggr__o_orderdate;
hipMalloc(&d_aggr__o_orderdate, sizeof(DBDateType) * COUNT63411b13d830);
hipMemset(d_aggr__o_orderdate, 0, sizeof(DBDateType) * COUNT63411b13d830);
DBI32Type* d_KEY_63411b13d830lineitem__l_orderkey;
hipMalloc(&d_KEY_63411b13d830lineitem__l_orderkey, sizeof(DBI32Type) * COUNT63411b13d830);
hipMemset(d_KEY_63411b13d830lineitem__l_orderkey, 0, sizeof(DBI32Type) * COUNT63411b13d830);
main_63411b18bef0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_63411b17fc20, d_HT_63411b13d830.ref(cuco::find), d_HT_63411b17fc20.ref(cuco::find), d_KEY_63411b13d830lineitem__l_orderkey, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, lineitem_size, d_orders__o_orderdate, d_orders__o_shippriority);
//Materialize count
uint64_t* d_COUNT63411b11e300;
hipMalloc(&d_COUNT63411b11e300, sizeof(uint64_t));
hipMemset(d_COUNT63411b11e300, 0, sizeof(uint64_t));
count_63411b196f70<<<std::ceil((float)COUNT63411b13d830/32.), 32>>>(d_COUNT63411b11e300, COUNT63411b13d830);
uint64_t COUNT63411b11e300;
hipMemcpy(&COUNT63411b11e300, d_COUNT63411b11e300, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT63411b11e300);
//Materialize buffers
auto MAT63411b11e300lineitem__l_orderkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT63411b11e300);
DBI32Type* d_MAT63411b11e300lineitem__l_orderkey;
hipMalloc(&d_MAT63411b11e300lineitem__l_orderkey, sizeof(DBI32Type) * COUNT63411b11e300);
auto MAT63411b11e300aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT63411b11e300);
DBDecimalType* d_MAT63411b11e300aggr0__tmp_attr0;
hipMalloc(&d_MAT63411b11e300aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT63411b11e300);
auto MAT63411b11e300aggr__o_orderdate = (DBDateType*)malloc(sizeof(DBDateType) * COUNT63411b11e300);
DBDateType* d_MAT63411b11e300aggr__o_orderdate;
hipMalloc(&d_MAT63411b11e300aggr__o_orderdate, sizeof(DBDateType) * COUNT63411b11e300);
auto MAT63411b11e300aggr__o_shippriority = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT63411b11e300);
DBI32Type* d_MAT63411b11e300aggr__o_shippriority;
hipMalloc(&d_MAT63411b11e300aggr__o_shippriority, sizeof(DBI32Type) * COUNT63411b11e300);
main_63411b196f70<<<std::ceil((float)COUNT63411b13d830/32.), 32>>>(COUNT63411b13d830, d_MAT63411b11e300aggr0__tmp_attr0, d_MAT63411b11e300aggr__o_orderdate, d_MAT63411b11e300aggr__o_shippriority, d_MAT63411b11e300lineitem__l_orderkey, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_KEY_63411b13d830lineitem__l_orderkey);
hipMemcpy(MAT63411b11e300lineitem__l_orderkey, d_MAT63411b11e300lineitem__l_orderkey, sizeof(DBI32Type) * COUNT63411b11e300, hipMemcpyDeviceToHost);
hipMemcpy(MAT63411b11e300aggr0__tmp_attr0, d_MAT63411b11e300aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT63411b11e300, hipMemcpyDeviceToHost);
hipMemcpy(MAT63411b11e300aggr__o_orderdate, d_MAT63411b11e300aggr__o_orderdate, sizeof(DBDateType) * COUNT63411b11e300, hipMemcpyDeviceToHost);
hipMemcpy(MAT63411b11e300aggr__o_shippriority, d_MAT63411b11e300aggr__o_shippriority, sizeof(DBI32Type) * COUNT63411b11e300, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT63411b11e300; i++) { std::cout << MAT63411b11e300lineitem__l_orderkey[i] << "\t";
std::cout << MAT63411b11e300aggr0__tmp_attr0[i] << "\t";
std::cout << MAT63411b11e300aggr__o_orderdate[i] << "\t";
std::cout << MAT63411b11e300aggr__o_shippriority[i] << "\t";
std::cout << std::endl; }
}