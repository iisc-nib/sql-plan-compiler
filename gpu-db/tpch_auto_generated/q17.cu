#include "hip/hip_runtime.h"
#include "utils.h"

#include <cuco/static_map.cuh>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <chrono>

namespace cg = cooperative_groups;

__global__ void pipeline_1(int32_t *l_partkey, int32_t *p_partkey, int8_t *p_container, int8_t *p_brand, size_t part_size, int64_t *B0_idx)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= part_size)
        return;
    int8_t reg_p_container = p_container[tid];
    if (!(reg_p_container == 17))
        return;
    int8_t reg_p_brand = p_brand[tid];
    if (!(reg_p_brand == 7))
        return;
    int32_t reg_p_partkey = p_partkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_p_partkey) << 0);
    atomicAdd((int *)B0_idx, 1);
}

template <typename TY_HT0_I, typename TY_HT0_F>
__global__ void pipeline_0(int32_t *l_partkey, int8_t *p_brand, int32_t *p_partkey, int8_t *p_container, TY_HT0_I HT0_I, TY_HT0_F HT0_F, int64_t *B0_part, size_t part_size, int64_t *B0_idx)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= part_size)
        return;
    int8_t reg_p_container = p_container[tid];
    if (!(reg_p_container ==17))
        return;
    int8_t reg_p_brand = p_brand[tid];
    if (!(reg_p_brand == 7))
        return;
    int32_t reg_p_partkey = p_partkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_p_partkey) << 0);
    auto reg_B0_idx = atomicAdd((int *)B0_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT0_I.insert(thread, cuco::pair{key0, reg_B0_idx});
    B0_part[reg_B0_idx] = tid;
}

template <typename TY_HT0_I, typename TY_HT0_F, typename TY_HT1_I, typename TY_HT1_F>
__global__ void pipeline_2(int32_t *l_partkey, TY_HT0_I HT0_I, TY_HT0_F HT0_F, TY_HT1_I HT1_I, TY_HT1_F HT1_F, int64_t *B0_part, size_t lineitem_size)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= lineitem_size)
        return;
    int32_t reg_l_partkey = l_partkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_l_partkey) << 0);
    auto slot0 = HT0_F.find(key0);
    if (slot0 == HT0_F.end())
        return;
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_l_partkey) << 0);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT1_I.insert(thread, cuco::pair{key1, 1});
}

template <typename TY_HT0_I, typename TY_HT0_F, typename TY_HT1_I, typename TY_HT1_F>
__global__ void pipeline_3(int32_t *l_partkey, int64_t *l_quantity, int64_t *sum_quantity, int32_t *agg_partkey, int64_t *agg_count, TY_HT0_I HT0_I, TY_HT0_F HT0_F, TY_HT1_I HT1_I, TY_HT1_F HT1_F, int64_t *B0_part, size_t lineitem_size)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= lineitem_size)
        return;
    int32_t reg_l_partkey = l_partkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_l_partkey) << 0);
    auto slot0 = HT0_F.find(key0);
    if (slot0 == HT0_F.end())
        return;
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_l_partkey) << 0);
    auto slot1 = HT1_F.find(key1);
    int64_t reg_l_quantity = l_quantity[tid];
    agg_partkey[slot1->second] = reg_l_partkey;
    aggregate_sum(&(sum_quantity[slot1->second]), reg_l_quantity);
    aggregate_sum(&(agg_count[slot1->second]), 1);
}

__global__ void pipeline_5(int32_t *l_partkey, int64_t *sum_quantity, int64_t *agg_count, int32_t *agg_partkey, int64_t *B2_idx, size_t agg_size)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= agg_size)
        return;
    int64_t reg_sum_quantity = sum_quantity[tid];
    int64_t reg_agg_count = agg_count[tid];
    int32_t reg_agg_partkey = agg_partkey[tid];
    int64_t key2 = 0;
    key2 |= (((int64_t)reg_agg_partkey) << 0);
    atomicAdd((int *)B2_idx, 1);
}

template <typename TY_HT2_I, typename TY_HT2_F>
__global__ void pipeline_4(int32_t *l_partkey, int64_t *agg_count, int64_t *sum_quantity, int32_t *agg_partkey, TY_HT2_I HT2_I, TY_HT2_F HT2_F, int64_t *B2_idx, size_t agg_size, int64_t *B2_agg)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= agg_size)
        return;
    int64_t reg_sum_quantity = sum_quantity[tid];
    int64_t reg_agg_count = agg_count[tid];
    int32_t reg_agg_partkey = agg_partkey[tid];
    int64_t key2 = 0;
    double reg_avg_qty = 0.2 * ((double)reg_sum_quantity / (double)reg_agg_count);
    key2 |= (((int64_t)reg_agg_partkey) << 0);
    auto reg_B2_idx = atomicAdd((int *)B2_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT2_I.insert(thread, cuco::pair{key2, reg_B2_idx});
    B2_agg[reg_B2_idx] = tid;
}

template <typename TY_HT2_I, typename TY_HT2_F, typename TY_HT3_I, typename TY_HT3_F>
__global__ void pipeline_6(int32_t *l_partkey, int64_t *l_quantity, int64_t *sum_quantity, int64_t *agg_count, TY_HT2_I HT2_I, TY_HT2_F HT2_F, TY_HT3_I HT3_I, TY_HT3_F HT3_F, size_t lineitem_size, int64_t *B2_agg)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= lineitem_size)
        return;
    int32_t reg_l_partkey = l_partkey[tid];
    int64_t reg_l_quantity = l_quantity[tid];
    int64_t key2 = 0;
    key2 |= (((int64_t)reg_l_partkey) << 0);
    auto slot2 = HT2_F.find(key2);
    if (slot2 == HT2_F.end())
        return;
    // check if reg_l_quantity < sum_quantity[B2_agg[slot2->second]]
    int64_t reg_sum_quantity = sum_quantity[B2_agg[slot2->second]];
    int64_t reg_agg_count = agg_count[B2_agg[slot2->second]];
    double reg_avg_qty = 0.2 * ((double)reg_sum_quantity / (double)reg_agg_count);
    if (!((double)reg_l_quantity < reg_avg_qty))
        return;
    int64_t key3 = 0;
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT3_I.insert(thread, cuco::pair{key3, 1});
}

template <typename TY_HT2_I, typename TY_HT2_F, typename TY_HT3_I, typename TY_HT3_F>
__global__ void pipeline_7(int32_t *l_partkey, int64_t *l_quantity, int64_t *sum_quantity, int64_t *agg_count, double *l_extendedprice, double *agg2_extendedprice, TY_HT2_I HT2_I, TY_HT2_F HT2_F, TY_HT3_I HT3_I, TY_HT3_F HT3_F, size_t lineitem_size, int64_t *B2_agg)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= lineitem_size)
        return;
    int32_t reg_l_partkey = l_partkey[tid];
    int64_t reg_l_quantity = l_quantity[tid];
    int64_t key2 = 0;
    key2 |= (((int64_t)reg_l_partkey) << 0);
    auto slot2 = HT2_F.find(key2);
    if (slot2 == HT2_F.end())
        return;

    int64_t reg_sum_quantity = sum_quantity[B2_agg[slot2->second]];
    int64_t reg_agg_count = agg_count[B2_agg[slot2->second]];
    double reg_avg_qty = 0.2 * ((double)reg_sum_quantity / (double)reg_agg_count);
    if (!((double)reg_l_quantity < reg_avg_qty))
        return;
    int64_t key3 = 0;
    auto slot3 = HT3_F.find(key3);
    double reg_l_extendedprice = l_extendedprice[tid];
    aggregate_sum(&(agg2_extendedprice[slot3->second]), reg_l_extendedprice);
}

void control(
    int32_t *l_partkey,
    double *l_extendedprice,
    int64_t *l_quantity,
    int8_t *p_brand,
    int8_t *p_container,
    int32_t *p_partkey,
    size_t lineitem_size,
    size_t part_size)
{
    std::cout << "HashJoin pipelines stats: \n";
    auto start_0 = std::chrono::high_resolution_clock::now();
    int32_t *d_l_partkey;

    hipMalloc(&d_l_partkey, sizeof(int32_t) * lineitem_size);

    hipMemcpy(d_l_partkey, l_partkey, sizeof(int32_t) * lineitem_size, hipMemcpyHostToDevice);

    int8_t *d_p_brand;

    hipMalloc(&d_p_brand, sizeof(int8_t) * part_size);

    hipMemcpy(d_p_brand, p_brand, sizeof(int8_t) * part_size, hipMemcpyHostToDevice);

    int32_t *d_p_partkey;

    hipMalloc(&d_p_partkey, sizeof(int32_t) * part_size);

    hipMemcpy(d_p_partkey, p_partkey, sizeof(int32_t) * part_size, hipMemcpyHostToDevice);

    int8_t *d_p_container;

    hipMalloc(&d_p_container, sizeof(int8_t) * part_size);

    hipMemcpy(d_p_container, p_container, sizeof(int8_t) * part_size, hipMemcpyHostToDevice);

    auto stop_0 = std::chrono::high_resolution_clock::now();
    auto duration_0 = std::chrono::duration_cast<std::chrono::milliseconds>(stop_0 - start_0);
    std::cout << "\tAlloc and init: " << duration_0.count() << " ms.\n";
    int64_t *B0_part;
    int64_t *B0_idx;
    hipMalloc(&B0_idx, sizeof(int64_t));
    hipMemset(B0_idx, 0, sizeof(int64_t));
    pipeline_1<<<std::ceil((float)part_size / (float)32), 32>>>(d_l_partkey, d_p_partkey, d_p_container, d_p_brand, part_size, B0_idx);

    int64_t h_B0_idx;
    hipMemcpy(&h_B0_idx, B0_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B0_idx, 0, sizeof(int64_t));
    hipMalloc(&B0_part, sizeof(int64_t) * h_B0_idx);
    auto HT0 = cuco::static_map{h_B0_idx * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT0_F = HT0.ref(cuco::find);

    auto d_HT0_I = HT0.ref(cuco::insert);

    pipeline_0<<<std::ceil((float)part_size / (float)32), 32>>>(d_l_partkey, d_p_brand, d_p_partkey, d_p_container, d_HT0_I, d_HT0_F, B0_part, part_size, B0_idx);

    std::cout << "Aggregation pipelines stats: \n";
    auto start_3 = std::chrono::high_resolution_clock::now();
    int64_t *d_l_quantity;

    hipMalloc(&d_l_quantity, sizeof(int64_t) * lineitem_size);

    hipMemcpy(d_l_quantity, l_quantity, sizeof(int64_t) * lineitem_size, hipMemcpyHostToDevice);

    auto stop_3 = std::chrono::high_resolution_clock::now();
    auto duration_3 = std::chrono::duration_cast<std::chrono::milliseconds>(stop_3 - start_3);
    std::cout << "\tAlloc and init: " << duration_3.count() << " ms.\n";
    int64_t *d_sum_quantity;

    int32_t *d_agg_partkey;

    int64_t *d_agg_count;

    auto HT1 = cuco::static_map{lineitem_size * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT1_F = HT1.ref(cuco::find);

    auto d_HT1_I = HT1.ref(cuco::insert);

    pipeline_2<<<std::ceil((float)lineitem_size / (float)32), 32>>>(d_l_partkey, d_HT0_I, d_HT0_F, d_HT1_I, d_HT1_F, B0_part, lineitem_size);

    auto HT1_size = HT1.size();

    hipMalloc(&d_sum_quantity, sizeof(int64_t) * HT1_size);

    hipMalloc(&d_agg_partkey, sizeof(int32_t) * HT1_size);

    hipMalloc(&d_agg_count, sizeof(int64_t) * HT1_size);

    hipMemset(d_sum_quantity, 0, sizeof(int64_t) * HT1_size);

    hipMemset(d_agg_partkey, 0, sizeof(int32_t) * HT1_size);

    hipMemset(d_agg_count, 0, sizeof(int64_t) * HT1_size);

    thrust::device_vector<int64_t> keys_1(HT1_size), vals_1(HT1_size);
    HT1.retrieve_all(keys_1.begin(), vals_1.begin());
    thrust::host_vector<int64_t> h_keys_1(HT1_size);
    thrust::copy(keys_1.begin(), keys_1.end(), h_keys_1.begin());
    thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_1(HT1_size);
    for (int i = 0; i < HT1_size; i++)
    {
        actual_dict_1[i] = cuco::make_pair(h_keys_1[i], i);
    }
    HT1.clear();
    HT1.insert(actual_dict_1.begin(), actual_dict_1.end());
    pipeline_3<<<std::ceil((float)lineitem_size / (float)32), 32>>>(d_l_partkey, d_l_quantity, d_sum_quantity, d_agg_partkey, d_agg_count, d_HT0_I, d_HT0_F, d_HT1_I, d_HT1_F, B0_part, lineitem_size);

    size_t agg_size = HT1_size;

    std::cout << "First aggregation size: " << agg_size << std::endl;
    int64_t *B2_agg;
    int64_t *B2_idx;
    hipMalloc(&B2_idx, sizeof(int64_t));
    hipMemset(B2_idx, 0, sizeof(int64_t));
    pipeline_5<<<std::ceil((float)agg_size / (float)32), 32>>>(d_l_partkey, d_sum_quantity, d_agg_count, d_agg_partkey, B2_idx, agg_size);

    int64_t h_B2_idx;
    hipMemcpy(&h_B2_idx, B2_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B2_idx, 0, sizeof(int64_t));
    hipMalloc(&B2_agg, sizeof(int64_t) * h_B2_idx);
    auto HT2 = cuco::static_map{h_B2_idx * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT2_F = HT2.ref(cuco::find);

    auto d_HT2_I = HT2.ref(cuco::insert);

    pipeline_4<<<std::ceil((float)agg_size / (float)32), 32>>>(d_l_partkey, d_agg_count, d_sum_quantity, d_agg_partkey, d_HT2_I, d_HT2_F, B2_idx, agg_size, B2_agg);

    std::cout << "Aggregation pipelines stats: \n";
    auto start_7 = std::chrono::high_resolution_clock::now();
    double *d_l_extendedprice;

    hipMalloc(&d_l_extendedprice, sizeof(double) * lineitem_size);

    hipMemcpy(d_l_extendedprice, l_extendedprice, sizeof(double) * lineitem_size, hipMemcpyHostToDevice);

    auto stop_7 = std::chrono::high_resolution_clock::now();
    auto duration_7 = std::chrono::duration_cast<std::chrono::milliseconds>(stop_7 - start_7);
    std::cout << "\tAlloc and init: " << duration_7.count() << " ms.\n";
    double *d_agg2_extendedprice;

    auto HT3 = cuco::static_map{lineitem_size * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT3_F = HT3.ref(cuco::find);

    auto d_HT3_I = HT3.ref(cuco::insert);

    pipeline_6<<<std::ceil((float)lineitem_size / (float)32), 32>>>(d_l_partkey, d_l_quantity, d_sum_quantity, d_agg_count, d_HT2_I, d_HT2_F, d_HT3_I, d_HT3_F, lineitem_size, B2_agg);

    auto HT3_size = HT3.size();

    hipMalloc(&d_agg2_extendedprice, sizeof(double) * HT3_size);

    hipMemset(d_agg2_extendedprice, 0, sizeof(double) * HT3_size);

    thrust::device_vector<int64_t> keys_3(HT3_size), vals_3(HT3_size);
    HT3.retrieve_all(keys_3.begin(), vals_3.begin());
    thrust::host_vector<int64_t> h_keys_3(HT3_size);
    thrust::copy(keys_3.begin(), keys_3.end(), h_keys_3.begin());
    thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_3(HT3_size);
    for (int i = 0; i < HT3_size; i++)
    {
        actual_dict_3[i] = cuco::make_pair(h_keys_3[i], i);
    }
    HT3.clear();
    HT3.insert(actual_dict_3.begin(), actual_dict_3.end());
    pipeline_7<<<std::ceil((float)lineitem_size / (float)32), 32>>>(d_l_partkey, d_l_quantity, d_sum_quantity, d_agg_count, d_l_extendedprice, d_agg2_extendedprice, d_HT2_I, d_HT2_F, d_HT3_I, d_HT3_F, lineitem_size, B2_agg);

    size_t agg2_size = HT3_size;
    double *p_agg2_extendedprice = (double *)malloc(sizeof(double) * agg2_size);
    hipMemcpy(p_agg2_extendedprice, d_agg2_extendedprice, sizeof(double) * agg2_size, hipMemcpyDeviceToHost);
    for (int i = 0; i < agg2_size; i++)
    {
        std::cout << p_agg2_extendedprice[i] / 7.0 << "\t";
        std::cout << std::endl;
    }
}
int main(int argc, const char **argv)
{
    std::string dbDir = getDataDir(argv, argc);
    std::string lineitem_file = dbDir + "lineitem.parquet";
    auto lineitem_table = getArrowTable(lineitem_file);
    size_t lineitem_size = lineitem_table->num_rows();
    std::string part_file = dbDir + "part.parquet";
    auto part_table = getArrowTable(part_file);
    size_t part_size = part_table->num_rows();
    auto l_partkey = read_column_typecasted<int32_t>(lineitem_table, "l_partkey");
    auto l_extendedprice = read_column<double>(lineitem_table, "l_extendedprice");
    auto l_quantity = read_column<int64_t>(lineitem_table, "l_quantity");
    StringDictEncodedColumn *p_brand = read_string_dict_encoded_column(part_table, "p_brand");
    StringDictEncodedColumn *p_container = read_string_dict_encoded_column(part_table, "p_container");
    auto p_partkey = read_column_typecasted<int32_t>(part_table, "p_partkey");
    control(l_partkey.data(), l_extendedprice.data(), l_quantity.data(), p_brand->column, p_container->column, p_partkey.data(), lineitem_size, part_size);
}
