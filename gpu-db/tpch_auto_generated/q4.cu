#include "hip/hip_runtime.h"
#include "utils.h"

#include <cuco/static_map.cuh>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
namespace cg = cooperative_groups;

__global__ void pipeline_1(int32_t *o_orderdate, int32_t *o_orderkey, int32_t *l_orderkey, size_t orders_size, int64_t *B0_idx)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= orders_size)
        return;
    int32_t reg_o_orderdate = o_orderdate[tid];
    if (!(reg_o_orderdate >= 8582))
        return;
    if (!(reg_o_orderdate < 8674))
        return;
    int32_t reg_o_orderkey = o_orderkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_o_orderkey) << 0);
    atomicAdd((int *)B0_idx, 1);
}

template <typename TY_HT0_I, typename TY_HT0_F>
__global__ void pipeline_0(int32_t *o_orderdate, int32_t *o_orderkey, int32_t *l_orderkey, TY_HT0_I HT0_I, TY_HT0_F HT0_F, int64_t *B0_orders, size_t orders_size, int64_t *B0_idx)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= orders_size)
        return;
    int32_t reg_o_orderdate = o_orderdate[tid];
    if (!(reg_o_orderdate >= 8582))
        return;
    if (!(reg_o_orderdate < 8674))
        return;
    int32_t reg_o_orderkey = o_orderkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_o_orderkey) << 0);
    auto reg_B0_idx = atomicAdd((int *)B0_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT0_I.insert(thread, cuco::pair{key0, reg_B0_idx});
    B0_orders[reg_B0_idx] = tid;
}

template <typename TY_HT0_I, typename TY_HT0_F, typename TY_HT1_I, typename TY_HT1_F>
__global__ void pipeline_2(int32_t *l_orderkey, int32_t *l_commitdate, int32_t *l_receiptdate, int8_t *o_orderpriority, TY_HT0_I HT0_I, TY_HT0_F HT0_F, TY_HT1_I HT1_I, TY_HT1_F HT1_F, size_t lineitem_size, int64_t *B0_orders)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= lineitem_size)
        return;
    int32_t reg_l_commitdate = l_commitdate[tid];
    if (!(reg_l_commitdate < l_receiptdate[tid]))
        return;
    int32_t reg_l_orderkey = l_orderkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_l_orderkey) << 0);
    auto slot0 = HT0_F.find(key0);
    if (slot0 == HT0_F.end())
        return;
    int8_t reg_o_orderpriority = o_orderpriority[B0_orders[slot0->second]];
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_o_orderpriority) << 0);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT1_I.insert(thread, cuco::pair{key1, 1});
}

template <typename TY_HT0_I, typename TY_HT0_F, typename TY_HT1_I, typename TY_HT1_F>
__global__ void pipeline_3(int32_t *l_orderkey, int32_t *l_commitdate, int32_t *l_receiptdate, int8_t *o_orderpriority, int8_t *agg_op, int64_t *agg_count, TY_HT0_I HT0_I, TY_HT0_F HT0_F, TY_HT1_I HT1_I, TY_HT1_F HT1_F, size_t lineitem_size, int64_t *B0_orders)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= lineitem_size)
        return;
    int32_t reg_l_commitdate = l_commitdate[tid];
    if (!(reg_l_commitdate < l_receiptdate[tid]))
        return;
    int32_t reg_l_orderkey = l_orderkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_l_orderkey) << 0);
    auto slot0 = HT0_F.find(key0);
    if (slot0 == HT0_F.end())
        return;
    int8_t reg_o_orderpriority = o_orderpriority[B0_orders[slot0->second]];
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_o_orderpriority) << 0);
    auto slot1 = HT1_F.find(key1);
    agg_op[slot1->second] = reg_o_orderpriority;
    aggregate_sum(&(agg_count[slot1->second]), 1);
}

void control(
    int32_t *l_commitdate,
    int32_t *l_receiptdate,
    int32_t *l_orderkey,
    int32_t *o_orderdate,
    int32_t *o_orderkey,
    int8_t *o_orderpriority,
    size_t lineitem_size,
    size_t orders_size)
{
    int32_t *d_o_orderdate;

    hipMalloc(&d_o_orderdate, sizeof(int32_t) * orders_size);

    hipMemcpy(d_o_orderdate, o_orderdate, sizeof(int32_t) * orders_size, hipMemcpyHostToDevice);

    int32_t *d_o_orderkey;

    hipMalloc(&d_o_orderkey, sizeof(int32_t) * orders_size);

    hipMemcpy(d_o_orderkey, o_orderkey, sizeof(int32_t) * orders_size, hipMemcpyHostToDevice);

    int32_t *d_l_orderkey;

    hipMalloc(&d_l_orderkey, sizeof(int32_t) * lineitem_size);

    hipMemcpy(d_l_orderkey, l_orderkey, sizeof(int32_t) * lineitem_size, hipMemcpyHostToDevice);

    int64_t *B0_orders;
    int64_t *B0_idx;
    hipMalloc(&B0_idx, sizeof(int64_t));
    hipMemset(B0_idx, 0, sizeof(int64_t));
    pipeline_1<<<std::ceil((float)orders_size / (float)32), 32>>>(d_o_orderdate, d_o_orderkey, d_l_orderkey, orders_size, B0_idx);

    int64_t h_B0_idx;
    hipMemcpy(&h_B0_idx, B0_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B0_idx, 0, sizeof(int64_t));
    hipMalloc(&B0_orders, sizeof(int64_t) * h_B0_idx);
    auto HT0 = cuco::static_map{h_B0_idx * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT0_F = HT0.ref(cuco::find);

    auto d_HT0_I = HT0.ref(cuco::insert);

    pipeline_0<<<std::ceil((float)orders_size / (float)32), 32>>>(d_o_orderdate, d_o_orderkey, d_l_orderkey, d_HT0_I, d_HT0_F, B0_orders, orders_size, B0_idx);

    int32_t *d_l_commitdate;

    hipMalloc(&d_l_commitdate, sizeof(int32_t) * lineitem_size);

    hipMemcpy(d_l_commitdate, l_commitdate, sizeof(int32_t) * lineitem_size, hipMemcpyHostToDevice);

    int32_t *d_l_receiptdate;

    hipMalloc(&d_l_receiptdate, sizeof(int32_t) * lineitem_size);

    hipMemcpy(d_l_receiptdate, l_receiptdate, sizeof(int32_t) * lineitem_size, hipMemcpyHostToDevice);


    int8_t *d_o_orderpriority;

    hipMalloc(&d_o_orderpriority, sizeof(int8_t) * orders_size);

    hipMemcpy(d_o_orderpriority, o_orderpriority, sizeof(int8_t) * orders_size, hipMemcpyHostToDevice);

    int8_t *d_agg_op;

    int64_t *d_agg_count;

    auto HT1 = cuco::static_map{lineitem_size * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT1_F = HT1.ref(cuco::find);

    auto d_HT1_I = HT1.ref(cuco::insert);

    pipeline_2<<<std::ceil((float)lineitem_size / (float)32), 32>>>(d_l_orderkey, d_l_commitdate, d_l_receiptdate, d_o_orderpriority, d_HT0_I, d_HT0_F, d_HT1_I, d_HT1_F, lineitem_size, B0_orders);

    auto HT1_size = HT1.size();

    hipMalloc(&d_agg_op, sizeof(int8_t) * HT1_size);

    hipMalloc(&d_agg_count, sizeof(int64_t) * HT1_size);

    hipMemset(d_agg_op, 0, sizeof(int8_t) * HT1_size);

    hipMemset(d_agg_count, 0, sizeof(int64_t) * HT1_size);

    thrust::device_vector<int64_t> keys_1(HT1_size), vals_1(HT1_size);
    HT1.retrieve_all(keys_1.begin(), vals_1.begin());
    thrust::host_vector<int64_t> h_keys_1(HT1_size);
    thrust::copy(keys_1.begin(), keys_1.end(), h_keys_1.begin());
    thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_1(HT1_size);
    for (int i = 0; i < HT1_size; i++)
    {
        actual_dict_1[i] = cuco::make_pair(h_keys_1[i], i);
    }
    HT1.clear();
    HT1.insert(actual_dict_1.begin(), actual_dict_1.end());
    pipeline_3<<<std::ceil((float)lineitem_size / (float)32), 32>>>(d_l_orderkey, d_l_commitdate, d_l_receiptdate, d_o_orderpriority, d_agg_op, d_agg_count, d_HT0_I, d_HT0_F, d_HT1_I, d_HT1_F, lineitem_size, B0_orders);

    size_t agg_size = HT1_size;
    int8_t *p_agg_op = (int8_t *)malloc(sizeof(int8_t) * agg_size);
    hipMemcpy(p_agg_op, d_agg_op, sizeof(int8_t) * agg_size, hipMemcpyDeviceToHost);
    int64_t *p_agg_count = (int64_t *)malloc(sizeof(int64_t) * agg_size);
    hipMemcpy(p_agg_count, d_agg_count, sizeof(int64_t) * agg_size, hipMemcpyDeviceToHost);
    for (int i = 0; i < agg_size; i++)
    {
        std::cout << (int)p_agg_op[i] << "\t";
        std::cout << p_agg_count[i] << "\t";
        std::cout << std::endl;
    }
}

int main(int argc, const char **argv)
{
    std::string dbDir = getDataDir(argv, argc);
    std::string lineitem_file = dbDir + "lineitem.parquet";
    std::string orders_file = dbDir + "orders.parquet";

    auto lineitem_table = getArrowTable(lineitem_file);
    auto orders_table = getArrowTable(orders_file);
    size_t lineitem_size = lineitem_table->num_rows();
    size_t orders_size = orders_table->num_rows();

    auto l_commitdate = read_column<int32_t>(lineitem_table, "l_commitdate");
    auto l_receiptdate = read_column<int32_t>(lineitem_table, "l_receiptdate");
    auto l_orderkey = read_column_typecasted<int32_t>(lineitem_table, "l_orderkey");

    auto o_orderdate = read_column<int32_t>(orders_table, "o_orderdate");
    auto o_orderkey = read_column_typecasted<int32_t>(orders_table, "o_orderkey");
    StringDictEncodedColumn *o_orderpriority =
        read_string_dict_encoded_column(orders_table, "o_orderpriority");
    control(
        l_commitdate.data(),
        l_receiptdate.data(),
        l_orderkey.data(),
        o_orderdate.data(),
        o_orderkey.data(),
        o_orderpriority->column,
        lineitem_size,
        orders_size);
}