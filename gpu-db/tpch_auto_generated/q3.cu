#include "hip/hip_runtime.h"
#include "utils.h"

#include <cuco/static_map.cuh>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace cg = cooperative_groups;

__global__ void pipeline_1 (int8_t* c_mktsegment, int32_t* c_custkey, int64_t* B0_idx, size_t customer_size) {
int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid >= customer_size) return;
int8_t reg_c_mktsegment = c_mktsegment[tid];
if (!(reg_c_mktsegment == 0)) return;
int32_t reg_c_custkey = c_custkey[tid];
int64_t key0 = 0;
key0 |= (((int64_t)reg_c_custkey) << 0);
atomicAdd((int*)B0_idx, 1);
}

template <typename TY_HT0_I, typename TY_HT0_F>
__global__ void pipeline_0 (int8_t* c_mktsegment, int32_t* c_custkey, TY_HT0_I HT0_I, TY_HT0_F HT0_F, int64_t* B0_customer, int64_t* B0_idx, size_t customer_size) {
int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid >= customer_size) return;
int8_t reg_c_mktsegment = c_mktsegment[tid];
if (!(reg_c_mktsegment == 0)) return;
int32_t reg_c_custkey = c_custkey[tid];
int64_t key0 = 0;
key0 |= (((int64_t)reg_c_custkey) << 0);
auto reg_B0_idx = atomicAdd((int*)B0_idx, 1);
auto thread = cg::tiled_partition<1>(cg::this_thread_block());
HT0_I.insert(thread, cuco::pair{key0, reg_B0_idx});
B0_customer[reg_B0_idx] = tid;
}

template <typename TY_HT0_I, typename TY_HT0_F>
__global__ void pipeline_3 (int32_t* o_orderdate, int32_t* o_custkey, int32_t* o_orderkey, TY_HT0_I HT0_I, TY_HT0_F HT0_F, size_t orders_size, int64_t* B0_customer, int64_t* B1_idx) {
int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid >= orders_size) return;
int32_t reg_o_orderdate = o_orderdate[tid];
if (!(reg_o_orderdate < 9204)) return;
int32_t reg_o_custkey = o_custkey[tid];
int64_t key0 = 0;
key0 |= (((int64_t)reg_o_custkey) << 0);
auto slot0 = HT0_F.find(key0);
if (slot0 == HT0_F.end()) return;int32_t reg_o_orderkey = o_orderkey[tid];
int64_t key1 = 0;
key1 |= (((int64_t)reg_o_orderkey) << 0);
atomicAdd((int*)B1_idx, 1);
}

template <typename TY_HT0_I, typename TY_HT0_F, typename TY_HT1_I, typename TY_HT1_F>
__global__ void pipeline_2 (int32_t* o_orderdate, int32_t* o_custkey, int32_t* o_orderkey, TY_HT0_I HT0_I, TY_HT0_F HT0_F, TY_HT1_I HT1_I, TY_HT1_F HT1_F, int64_t* B1_orders, int64_t* B1_idx, int64_t* B1_customer, size_t orders_size, int64_t* B0_customer) {
int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid >= orders_size) return;
int32_t reg_o_orderdate = o_orderdate[tid];
if (!(reg_o_orderdate < 9204)) return;
int32_t reg_o_custkey = o_custkey[tid];
int64_t key0 = 0;
key0 |= (((int64_t)reg_o_custkey) << 0);
auto slot0 = HT0_F.find(key0);
if (slot0 == HT0_F.end()) return;int32_t reg_o_orderkey = o_orderkey[tid];
int64_t key1 = 0;
key1 |= (((int64_t)reg_o_orderkey) << 0);
auto reg_B1_idx = atomicAdd((int*)B1_idx, 1);
auto thread = cg::tiled_partition<1>(cg::this_thread_block());
HT1_I.insert(thread, cuco::pair{key1, reg_B1_idx});
B1_orders[reg_B1_idx] = tid;
B1_customer[reg_B1_idx] = B0_customer[slot0->second];
}

template <typename TY_HT1_I, typename TY_HT1_F, typename TY_HT2_I, typename TY_HT2_F>
__global__ void pipeline_4 (double* l_discount, int32_t* l_shipdate, int32_t* l_orderkey, double* l_extendedprice, TY_HT1_I HT1_I, TY_HT1_F HT1_F, TY_HT2_I HT2_I, TY_HT2_F HT2_F, size_t lineitem_size, int64_t* B1_orders, int64_t* B1_customer) {
int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid >= lineitem_size) return;
int32_t reg_l_shipdate = l_shipdate[tid];
if (!(reg_l_shipdate > 9204)) return;
int32_t reg_l_orderkey = l_orderkey[tid];
int64_t key1 = 0;
key1 |= (((int64_t)reg_l_orderkey) << 0);
auto slot1 = HT1_F.find(key1);
if (slot1 == HT1_F.end()) return;double reg_l_extendedprice = l_extendedprice[tid];
double reg_l_discount = l_discount[tid];
double reg_revenue = (reg_l_extendedprice * (1 - reg_l_discount));
int64_t key2 = 0;
key2 |= (((int64_t)reg_l_orderkey) << 0);
auto thread = cg::tiled_partition<1>(cg::this_thread_block());
HT2_I.insert(thread, cuco::pair{key2, 1});
}

template <typename TY_HT1_I, typename TY_HT1_F, typename TY_HT2_I, typename TY_HT2_F>
__global__ void pipeline_5 (int32_t* o_shippriority, int32_t* l_orderkey, int32_t* o_orderdate, double* l_extendedprice, int32_t* l_shipdate, double* l_discount, int32_t* agg_l_orderkey, int32_t* agg_o_shippriority, double* agg_l_discount, double* revenue, int32_t* agg_o_orderdate, TY_HT1_I HT1_I, TY_HT1_F HT1_F, TY_HT2_I HT2_I, TY_HT2_F HT2_F, size_t lineitem_size, int64_t* B1_orders, int64_t* B1_customer) {
int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid >= lineitem_size) return;
int32_t reg_l_shipdate = l_shipdate[tid];
if (!(reg_l_shipdate > 9204)) return;
int32_t reg_l_orderkey = l_orderkey[tid];
int64_t key1 = 0;
key1 |= (((int64_t)reg_l_orderkey) << 0);
auto slot1 = HT1_F.find(key1);
if (slot1 == HT1_F.end()) return;double reg_l_extendedprice = l_extendedprice[tid];
double reg_l_discount = l_discount[tid];
double reg_revenue = (reg_l_extendedprice * (1 - reg_l_discount));
int64_t key2 = 0;
key2 |= (((int64_t)reg_l_orderkey) << 0);
auto slot2 = HT2_F.find(key2);
int32_t reg_o_shippriority = o_shippriority[B1_orders[slot1->second]];
int32_t reg_o_orderdate = o_orderdate[B1_orders[slot1->second]];
agg_l_orderkey[slot2->second] = reg_l_orderkey;
aggregate_sum(&(agg_l_discount[slot2->second]), reg_l_discount);
agg_o_shippriority[slot2->second] = reg_o_shippriority;
aggregate_sum(&(revenue[slot2->second]), reg_revenue);
agg_o_orderdate[slot2->second] = reg_o_orderdate;
}

void control(
int32_t *c_custkey,
int8_t *c_mktsegment,
double *l_discount,
double *l_extendedprice,
int32_t *l_orderkey,
int32_t *l_shipdate,
int32_t *o_custkey,
int32_t *o_orderdate,
int32_t *o_orderkey,
int32_t *o_shippriority,
size_t customer_size,
size_t lineitem_size,
size_t orders_size
) {
int8_t* d_c_mktsegment;

hipMalloc(&d_c_mktsegment, sizeof(int8_t) * customer_size);

hipMemcpy(d_c_mktsegment, c_mktsegment, sizeof(int8_t) * customer_size, hipMemcpyHostToDevice);

int32_t* d_c_custkey;

hipMalloc(&d_c_custkey, sizeof(int32_t) * customer_size);

hipMemcpy(d_c_custkey, c_custkey, sizeof(int32_t) * customer_size, hipMemcpyHostToDevice);

int64_t* B0_customer;
int64_t* B0_idx;
hipMalloc(&B0_idx, sizeof(int64_t));
hipMemset(B0_idx, 0, sizeof(int64_t));
pipeline_1<<<std::ceil((float)customer_size/(float)32), 32>>>(d_c_mktsegment, d_c_custkey, B0_idx, customer_size);

int64_t h_B0_idx;
hipMemcpy(&h_B0_idx, B0_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
hipMemset(B0_idx, 0, sizeof(int64_t));
hipMalloc(&B0_customer, sizeof(int64_t) * h_B0_idx);
auto HT0 = cuco::static_map{ h_B0_idx * 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

auto d_HT0_F = HT0.ref(cuco::find);

auto d_HT0_I = HT0.ref(cuco::insert);

pipeline_0<<<std::ceil((float)customer_size/(float)32), 32>>>(d_c_mktsegment, d_c_custkey, d_HT0_I, d_HT0_F, B0_customer, B0_idx, customer_size);

int32_t* d_o_orderdate;

hipMalloc(&d_o_orderdate, sizeof(int32_t) * orders_size);

hipMemcpy(d_o_orderdate, o_orderdate, sizeof(int32_t) * orders_size, hipMemcpyHostToDevice);

int32_t* d_o_custkey;

hipMalloc(&d_o_custkey, sizeof(int32_t) * orders_size);

hipMemcpy(d_o_custkey, o_custkey, sizeof(int32_t) * orders_size, hipMemcpyHostToDevice);

int32_t* d_o_orderkey;

hipMalloc(&d_o_orderkey, sizeof(int32_t) * orders_size);

hipMemcpy(d_o_orderkey, o_orderkey, sizeof(int32_t) * orders_size, hipMemcpyHostToDevice);

int64_t* B1_orders;
int64_t* B1_customer;
int64_t* B1_idx;
hipMalloc(&B1_idx, sizeof(int64_t));
hipMemset(B1_idx, 0, sizeof(int64_t));
pipeline_3<<<std::ceil((float)orders_size/(float)32), 32>>>(d_o_orderdate, d_o_custkey, d_o_orderkey, d_HT0_I, d_HT0_F, orders_size, B0_customer, B1_idx);

int64_t h_B1_idx;
hipMemcpy(&h_B1_idx, B1_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
hipMemset(B1_idx, 0, sizeof(int64_t));
hipMalloc(&B1_orders, sizeof(int64_t) * h_B1_idx);
hipMalloc(&B1_customer, sizeof(int64_t) * h_B1_idx);
auto HT1 = cuco::static_map{ h_B1_idx * 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

auto d_HT1_F = HT1.ref(cuco::find);

auto d_HT1_I = HT1.ref(cuco::insert);

pipeline_2<<<std::ceil((float)orders_size/(float)32), 32>>>(d_o_orderdate, d_o_custkey, d_o_orderkey, d_HT0_I, d_HT0_F, d_HT1_I, d_HT1_F, B1_orders, B1_idx, B1_customer, orders_size, B0_customer);

int32_t* d_o_shippriority;

hipMalloc(&d_o_shippriority, sizeof(int32_t) * orders_size);

hipMemcpy(d_o_shippriority, o_shippriority, sizeof(int32_t) * orders_size, hipMemcpyHostToDevice);

int32_t* d_l_orderkey;

hipMalloc(&d_l_orderkey, sizeof(int32_t) * lineitem_size);

hipMemcpy(d_l_orderkey, l_orderkey, sizeof(int32_t) * lineitem_size, hipMemcpyHostToDevice);

double* d_l_extendedprice;

hipMalloc(&d_l_extendedprice, sizeof(double) * lineitem_size);

hipMemcpy(d_l_extendedprice, l_extendedprice, sizeof(double) * lineitem_size, hipMemcpyHostToDevice);

int32_t* d_l_shipdate;

hipMalloc(&d_l_shipdate, sizeof(int32_t) * lineitem_size);

hipMemcpy(d_l_shipdate, l_shipdate, sizeof(int32_t) * lineitem_size, hipMemcpyHostToDevice);

double* d_l_discount;

hipMalloc(&d_l_discount, sizeof(double) * lineitem_size);

hipMemcpy(d_l_discount, l_discount, sizeof(double) * lineitem_size, hipMemcpyHostToDevice);

int32_t* d_agg_l_orderkey;

int32_t* d_agg_o_shippriority;

double* d_agg_l_discount;

double* d_revenue;

int32_t* d_agg_o_orderdate;

auto HT2 = cuco::static_map{ lineitem_size * 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

auto d_HT2_F = HT2.ref(cuco::find);

auto d_HT2_I = HT2.ref(cuco::insert);

pipeline_4<<<std::ceil((float)lineitem_size/(float)32), 32>>>(d_l_discount, d_l_shipdate, d_l_orderkey, d_l_extendedprice, d_HT1_I, d_HT1_F, d_HT2_I, d_HT2_F, lineitem_size, B1_orders, B1_customer);

auto HT2_size = HT2.size();

hipMalloc(&d_agg_l_orderkey, sizeof(int32_t) * HT2_size);

hipMalloc(&d_agg_o_shippriority, sizeof(int32_t) * HT2_size);

hipMalloc(&d_agg_l_discount, sizeof(double) * HT2_size);

hipMalloc(&d_revenue, sizeof(double) * HT2_size);

hipMalloc(&d_agg_o_orderdate, sizeof(int32_t) * HT2_size);

hipMemset(d_agg_l_orderkey, 0, sizeof(int32_t) * HT2_size);

hipMemset(d_agg_o_shippriority, 0, sizeof(int32_t) * HT2_size);

hipMemset(d_agg_l_discount, 0, sizeof(double) * HT2_size);

hipMemset(d_revenue, 0, sizeof(double) * HT2_size);

hipMemset(d_agg_o_orderdate, 0, sizeof(int32_t) * HT2_size);

thrust::device_vector<int64_t> keys_2(HT2_size), vals_2(HT2_size);
HT2.retrieve_all(keys_2.begin(), vals_2.begin());
thrust::host_vector<int64_t> h_keys_2(HT2_size);
thrust::copy(keys_2.begin(), keys_2.end(), h_keys_2.begin());
thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_2(HT2_size);
for (int i=0; i < HT2_size; i++) {
actual_dict_2[i] = cuco::make_pair(h_keys_2[i], i);
}
HT2.clear();
HT2.insert(actual_dict_2.begin(), actual_dict_2.end());
pipeline_5<<<std::ceil((float)lineitem_size/(float)32), 32>>>(d_o_shippriority, d_l_orderkey, d_o_orderdate, d_l_extendedprice, d_l_shipdate, d_l_discount, d_agg_l_orderkey, d_agg_o_shippriority, d_agg_l_discount, d_revenue, d_agg_o_orderdate, d_HT1_I, d_HT1_F, d_HT2_I, d_HT2_F, lineitem_size, B1_orders, B1_customer);

size_t agg_size = HT2_size;
int32_t* p_agg_l_orderkey = (int32_t*)malloc(sizeof(int32_t) * agg_size);
hipMemcpy(p_agg_l_orderkey, d_agg_l_orderkey, sizeof(int32_t) * agg_size, hipMemcpyDeviceToHost);
double* p_revenue = (double*)malloc(sizeof(double) * agg_size);
hipMemcpy(p_revenue, d_revenue, sizeof(double) * agg_size, hipMemcpyDeviceToHost);
int32_t* p_agg_o_orderdate = (int32_t*)malloc(sizeof(int32_t) * agg_size);
hipMemcpy(p_agg_o_orderdate, d_agg_o_orderdate, sizeof(int32_t) * agg_size, hipMemcpyDeviceToHost);
int32_t* p_agg_o_shippriority = (int32_t*)malloc(sizeof(int32_t) * agg_size);
hipMemcpy(p_agg_o_shippriority, d_agg_o_shippriority, sizeof(int32_t) * agg_size, hipMemcpyDeviceToHost);
for (int i=0; i<agg_size; i++) {
std::cout << p_agg_l_orderkey[i] << "\t";
std::cout << p_revenue[i] << "\t";
std::cout << p_agg_o_orderdate[i] << "\t";
std::cout << p_agg_o_shippriority[i] << "\t";
std::cout << std::endl;
}
}


int main(int argc, const char** argv)
{
  std::string dbDir         = getDataDir(argv, argc);
  std::string lineitem_file = dbDir + "lineitem.parquet";
  std::string orders_file   = dbDir + "orders.parquet";
  std::string customer_file = dbDir + "customer.parquet";

  // auto lineitem_table  = getArrowTable(lineitem_file);
  auto orders_table    = getArrowTable(orders_file);
  auto customer_table  = getArrowTable(customer_file);
  auto lineitem_table  = getArrowTable(lineitem_file);
  size_t orders_size   = orders_table->num_rows();
  size_t customer_size = customer_table->num_rows();
  size_t lineitem_size = lineitem_table->num_rows();

  auto l_shipdate    = read_column<int32_t>(lineitem_table, "l_shipdate");
  auto l_discount    = read_column<double>(lineitem_table, "l_discount");
  auto l_extendedprice    = read_column<double>(lineitem_table, "l_extendedprice");
  auto l_orderkey      = read_column_typecasted<int32_t>(lineitem_table, "l_orderkey");

  auto o_custkey      = read_column_typecasted<int32_t>(orders_table, "o_custkey");
  auto o_orderdate    = read_column<int32_t>(orders_table, "o_orderdate");
  auto o_shippriority = read_column_typecasted<int32_t>(orders_table, "o_shippriority");
  auto o_orderkey     = read_column_typecasted<int32_t>(orders_table, "o_orderkey");

  StringDictEncodedColumn* c_mktsegment =
    read_string_dict_encoded_column(customer_table, "c_mktsegment");
  auto c_custkey      = read_column_typecasted<int32_t>(customer_table, "c_custkey");

  // for (auto p: c_mktsegment->dict) {
  //   std::cout << p.first << " " << (int)p.second << std::endl;
  // }
  control(
          c_custkey.data(),
          c_mktsegment->column,
          l_discount.data(),
          l_extendedprice.data(),
          l_orderkey.data(),
    l_shipdate.data(),
          o_custkey.data(),
          o_orderdate.data(),
          o_orderkey.data(),
          o_shippriority.data(),
          customer_size,
          lineitem_size,
          orders_size);
}