#include "hip/hip_runtime.h"
#include "utils.h"

#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace cg = cooperative_groups;

__global__ void pipeline_1(char *p_name, int64_t *p_name_offsets, int *p_name_sizes, int32_t *p_partkey, int32_t *l_partkey, size_t part_size, int64_t *B0_idx)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= part_size)
        return;
    if (!(like_operator(p_name + p_name_offsets[tid], p_name_sizes[tid], "green", 5)))
        return;
    int32_t reg_p_partkey = p_partkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_p_partkey) << 0);
    atomicAdd((int *)B0_idx, 1);
}

template <typename TY_HT0_I, typename TY_HT0_F>
__global__ void pipeline_0(char *p_name, int64_t *p_name_offsets, int *p_name_sizes, int32_t *l_partkey, int32_t *p_partkey, TY_HT0_I HT0_I, TY_HT0_F HT0_F, size_t part_size, int64_t *B0_idx, int64_t *B0_part)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= part_size)
        return;
    if (!(like_operator(p_name + p_name_offsets[tid], p_name_sizes[tid], "green", 5)))
        return;
    int32_t reg_p_partkey = p_partkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_p_partkey) << 0);
    auto reg_B0_idx = atomicAdd((int *)B0_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT0_I.insert(thread, cuco::pair{key0, reg_B0_idx});
    B0_part[reg_B0_idx] = tid;
}

template <typename TY_HT0_I, typename TY_HT0_F>
__global__ void pipeline_3(int32_t *l_partkey, int32_t *l_orderkey, TY_HT0_I HT0_I, TY_HT0_F HT0_F, size_t lineitem_size, int64_t *B0_part, int64_t *B1_idx)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= lineitem_size)
        return;
    int32_t reg_l_partkey = l_partkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_l_partkey) << 0);
    auto slot0 = HT0_F.find(key0);
    if (slot0 == HT0_F.end())
        return;
    int32_t reg_l_orderkey = l_orderkey[tid];
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_l_orderkey) << 0);
    atomicAdd((int *)B1_idx, 1);
}

template <typename TY_HT0_I, typename TY_HT0_F, typename TY_HT1_I, typename TY_HT1_F>
__global__ void pipeline_2(int32_t *l_partkey, int32_t *l_orderkey, TY_HT0_I HT0_I, TY_HT0_F HT0_F, TY_HT1_I HT1_I, TY_HT1_F HT1_F, size_t lineitem_size, int64_t *B1_lineitem, int64_t *B1_part, int64_t *B0_part, int64_t *B1_idx)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= lineitem_size)
        return;
    int32_t reg_l_partkey = l_partkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_l_partkey) << 0);
    auto slot0 = HT0_F.find(key0);
    if (slot0 == HT0_F.end())
        return;
    int32_t reg_l_orderkey = l_orderkey[tid];
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_l_orderkey) << 0);
    auto reg_B1_idx = atomicAdd((int *)B1_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT1_I.insert(thread, cuco::pair{key1, reg_B1_idx});
    B1_part[reg_B1_idx] = B0_part[slot0->second];
    B1_lineitem[reg_B1_idx] = tid;
}

template <typename TY_HT1_I, typename TY_HT1_F>
__global__ void pipeline_5(int32_t *l_partkey, int32_t *o_orderkey, int32_t *l_suppkey, TY_HT1_I HT1_I, TY_HT1_F HT1_F, int64_t *B1_part, size_t orders_size, int64_t *B4_idx, int64_t *B1_lineitem)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= orders_size)
        return;
    int32_t reg_o_orderkey = o_orderkey[tid];
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_o_orderkey) << 0);
    HT1_F.for_each(key1, [&] __device__(auto const slot1)
                   {
    auto const [slot1_key, slot1_val] = slot1;
    int32_t reg_l_partkey = l_partkey[B1_lineitem[slot1_val]];
    int32_t reg_l_suppkey = l_suppkey[B1_lineitem[slot1_val]];
    int64_t key4 = 0;
    key4 |= (((int64_t)reg_l_partkey) << 0);
    key4 |= (((int64_t)reg_l_suppkey) << 32);
    atomicAdd((int*)B4_idx, 1); });
}

template <typename TY_HT1_I, typename TY_HT1_F, typename TY_HT4_I, typename TY_HT4_F>
__global__ void pipeline_4(int32_t *l_partkey, int32_t *o_orderkey, int32_t *l_suppkey, TY_HT1_I HT1_I, TY_HT1_F HT1_F, TY_HT4_I HT4_I, TY_HT4_F HT4_F, int64_t *B4_idx, int64_t *B4_orders, int64_t *B1_lineitem, int64_t *B1_part, size_t orders_size, int64_t *B4_lineitem, int64_t *B4_part)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= orders_size)
        return;
    int32_t reg_o_orderkey = o_orderkey[tid];
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_o_orderkey) << 0);
    HT1_F.for_each(key1, [&] __device__(auto const slot1)
                   {
    auto const [slot1_key, slot1_val] = slot1;
    int32_t reg_l_partkey = l_partkey[B1_lineitem[slot1_val]];
    int32_t reg_l_suppkey = l_suppkey[B1_lineitem[slot1_val]];
    int64_t key4 = 0;
    key4 |= (((int64_t)reg_l_partkey) << 0);
    key4 |= (((int64_t)reg_l_suppkey) << 32);
    auto reg_B4_idx = atomicAdd((int*)B4_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT4_I.insert(thread, cuco::pair{key4, reg_B4_idx});
    B4_orders[reg_B4_idx] = tid;
    B4_part[reg_B4_idx] = B1_part[slot1_val];
    B4_lineitem[reg_B4_idx] = B1_lineitem[slot1_val]; });
}

__global__ void pipeline_7(int32_t *n_nationkey, int32_t *s_nationkey, int64_t *B2_idx, size_t nation_size)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= nation_size)
        return;
    int32_t reg_n_nationkey = n_nationkey[tid];
    int64_t key2 = 0;
    key2 |= (((int64_t)reg_n_nationkey) << 0);
    atomicAdd((int *)B2_idx, 1);
}

template <typename TY_HT2_I, typename TY_HT2_F>
__global__ void pipeline_6(int32_t *n_nationkey, int32_t *s_nationkey, TY_HT2_I HT2_I, TY_HT2_F HT2_F, int64_t *B2_idx, int64_t *B2_nation, size_t nation_size)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= nation_size)
        return;
    int32_t reg_n_nationkey = n_nationkey[tid];
    int64_t key2 = 0;
    key2 |= (((int64_t)reg_n_nationkey) << 0);
    auto reg_B2_idx = atomicAdd((int *)B2_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT2_I.insert(thread, cuco::pair{key2, reg_B2_idx});
    B2_nation[reg_B2_idx] = tid;
}

template <typename TY_HT2_I, typename TY_HT2_F>
__global__ void pipeline_9(int32_t *s_nationkey, int32_t *s_suppkey, int32_t *ps_suppkey, TY_HT2_I HT2_I, TY_HT2_F HT2_F, int64_t *B3_idx, size_t supplier_size, int64_t *B2_nation)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= supplier_size)
        return;
    int32_t reg_s_nationkey = s_nationkey[tid];
    int64_t key2 = 0;
    key2 |= (((int64_t)reg_s_nationkey) << 0);
    auto slot2 = HT2_F.find(key2);
    if (slot2 == HT2_F.end())
        return;
    int32_t reg_s_suppkey = s_suppkey[tid];
    int64_t key3 = 0;
    key3 |= (((int64_t)reg_s_suppkey) << 0);
    atomicAdd((int *)B3_idx, 1);
}

template <typename TY_HT2_I, typename TY_HT2_F, typename TY_HT3_I, typename TY_HT3_F>
__global__ void pipeline_8(int32_t *ps_suppkey, int32_t *s_suppkey, int32_t *s_nationkey, TY_HT2_I HT2_I, TY_HT2_F HT2_F, TY_HT3_I HT3_I, TY_HT3_F HT3_F, int64_t *B3_nation, int64_t *B3_supplier, size_t supplier_size, int64_t *B2_nation, int64_t *B3_idx)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= supplier_size)
        return;
    int32_t reg_s_nationkey = s_nationkey[tid];
    int64_t key2 = 0;
    key2 |= (((int64_t)reg_s_nationkey) << 0);
    auto slot2 = HT2_F.find(key2);
    if (slot2 == HT2_F.end())
        return;
    int32_t reg_s_suppkey = s_suppkey[tid];
    int64_t key3 = 0;
    key3 |= (((int64_t)reg_s_suppkey) << 0);
    auto reg_B3_idx = atomicAdd((int *)B3_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT3_I.insert(thread, cuco::pair{key3, reg_B3_idx});
    B3_supplier[reg_B3_idx] = tid;
    B3_nation[reg_B3_idx] = B2_nation[slot2->second];
}

template <typename TY_HT3_I, typename TY_HT3_F, typename TY_HT4_I, typename TY_HT4_F, typename TY_HT5_I, typename TY_HT5_F>
__global__ void pipeline_10(int32_t *ps_partkey, int32_t *ps_suppkey, double *l_extendedprice, double *l_discount, double *ps_supplycost, int8_t *n_name, int32_t *s_suppkey, int64_t *l_quantity, int32_t *o_orderdate, TY_HT3_I HT3_I, TY_HT3_F HT3_F, TY_HT4_I HT4_I, TY_HT4_F HT4_F, TY_HT5_I HT5_I, TY_HT5_F HT5_F, int64_t *B3_nation, int64_t *B3_supplier, int64_t *B4_orders, int64_t *B4_lineitem, size_t partsupp_size, int64_t *B4_part)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= partsupp_size)
        return;
    int32_t reg_ps_suppkey = ps_suppkey[tid];
    int64_t key3 = 0;
    key3 |= (((int64_t)reg_ps_suppkey) << 0);
    auto slot3 = HT3_F.find(key3);
    if (slot3 == HT3_F.end())
        return;
    int32_t reg_ps_partkey = ps_partkey[tid];
    int32_t reg_s_suppkey = s_suppkey[B3_supplier[slot3->second]];
    int64_t key4 = 0;
    key4 |= (((int64_t)reg_ps_partkey) << 0);
    key4 |= (((int64_t)reg_s_suppkey) << 32);
    HT4_F.for_each(key4, [&] __device__(auto const slot4)
                   {
    auto const [slot4_key, slot4_val] = slot4;
    double reg_l_extendedprice = l_extendedprice[B4_lineitem[slot4_val]];
    double reg_l_discount = l_discount[B4_lineitem[slot4_val]];
    double reg_ps_supplycost = ps_supplycost[tid];
    int64_t reg_l_quantity = l_quantity[B4_lineitem[slot4_val]];
    double reg_profit = ((reg_l_extendedprice * (1 - reg_l_discount))-(reg_ps_supplycost * reg_l_quantity));
    int8_t reg_n_name = n_name[B3_nation[slot3->second]];
    int32_t reg_o_orderdate = extract_year(o_orderdate[B4_orders[slot4_val]]);
    int64_t key5 = 0;
    key5 |= (((int64_t)reg_n_name) << 0);
    key5 |= (((int64_t)reg_o_orderdate) << 8);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT5_I.insert(thread, cuco::pair{key5, 1}); });
}

template <typename TY_HT3_I, typename TY_HT3_F, typename TY_HT4_I, typename TY_HT4_F, typename TY_HT5_I, typename TY_HT5_F>
__global__ void pipeline_11(int32_t *ps_partkey, int32_t *ps_suppkey, double *l_extendedprice, double *ps_supplycost, double *l_discount, int8_t *n_name, int32_t *s_suppkey, int64_t *l_quantity, int32_t *o_orderdate, int32_t *agg_o_orderdate, double *sum_profit, int8_t *agg_n_name, TY_HT3_I HT3_I, TY_HT3_F HT3_F, TY_HT4_I HT4_I, TY_HT4_F HT4_F, TY_HT5_I HT5_I, TY_HT5_F HT5_F, int64_t *B3_nation, int64_t *B3_supplier, int64_t *B4_orders, int64_t *B4_lineitem, size_t partsupp_size, int64_t *B4_part)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= partsupp_size)
        return;
    int32_t reg_ps_suppkey = ps_suppkey[tid];
    int64_t key3 = 0;
    key3 |= (((int64_t)reg_ps_suppkey) << 0);
    auto slot3 = HT3_F.find(key3);
    if (slot3 == HT3_F.end())
        return;
    int32_t reg_ps_partkey = ps_partkey[tid];
    int32_t reg_s_suppkey = s_suppkey[B3_supplier[slot3->second]];
    int64_t key4 = 0;
    key4 |= (((int64_t)reg_ps_partkey) << 0);
    key4 |= (((int64_t)reg_s_suppkey) << 32);
    HT4_F.for_each(key4, [&] __device__(auto const slot4)
                   {
    auto const [slot4_key, slot4_val] = slot4;
    double reg_l_extendedprice = l_extendedprice[B4_lineitem[slot4_val]];
    double reg_l_discount = l_discount[B4_lineitem[slot4_val]];
    double reg_ps_supplycost = ps_supplycost[tid];
    int64_t reg_l_quantity = l_quantity[B4_lineitem[slot4_val]];
    double reg_profit = ((reg_l_extendedprice * (1 - reg_l_discount))-(reg_ps_supplycost * reg_l_quantity));
    int8_t reg_n_name = n_name[B3_nation[slot3->second]];
    int32_t reg_o_orderdate = extract_year(o_orderdate[B4_orders[slot4_val]]);
    int64_t key5 = 0;
    key5 |= (((int64_t)reg_n_name) << 0);
    key5 |= (((int64_t)reg_o_orderdate) << 8);
    auto slot5 = HT5_F.find(key5);
    agg_n_name[slot5->second] = reg_n_name;
    agg_o_orderdate[slot5->second] = reg_o_orderdate;
    aggregate_sum(&(sum_profit[slot5->second]), reg_profit); });
}

void control(
    double *l_discount,
    double *l_extendedprice,
    int32_t *l_orderkey,
    int32_t *l_partkey,
    int64_t *l_quantity,
    int32_t *l_suppkey,
    int8_t *n_name,
    int32_t *n_nationkey,
    int32_t *o_orderdate,
    int32_t *o_orderkey,
    StringColumn *p_name,
    int32_t *p_partkey,
    int32_t *ps_partkey,
    int32_t *ps_suppkey,
    double *ps_supplycost,
    int32_t *s_nationkey,
    int32_t *s_suppkey,
    size_t lineitem_size,
    size_t nation_size,
    size_t orders_size,
    size_t part_size,
    size_t partsupp_size,
    size_t supplier_size)
{
    char *d_p_name;

    int64_t *d_p_name_offsets;

    int *d_p_name_sizes;

    hipMalloc(&d_p_name, sizeof(char) * (p_name->offsets[part_size - 1] + p_name->sizes[part_size - 1]));

    hipMemcpy(d_p_name, p_name->data, sizeof(char) * (p_name->offsets[part_size - 1] + p_name->sizes[part_size - 1]), hipMemcpyHostToDevice);

    hipMalloc(&d_p_name_offsets, sizeof(int64_t) * part_size);

    hipMemcpy(d_p_name_offsets, p_name->offsets, sizeof(int64_t) * part_size, hipMemcpyHostToDevice);

    hipMalloc(&d_p_name_sizes, sizeof(int) * part_size);

    hipMemcpy(d_p_name_sizes, p_name->sizes, sizeof(int) * part_size, hipMemcpyHostToDevice);

    int32_t *d_l_partkey;

    hipMalloc(&d_l_partkey, sizeof(int32_t) * lineitem_size);

    hipMemcpy(d_l_partkey, l_partkey, sizeof(int32_t) * lineitem_size, hipMemcpyHostToDevice);

    int32_t *d_p_partkey;

    hipMalloc(&d_p_partkey, sizeof(int32_t) * part_size);

    hipMemcpy(d_p_partkey, p_partkey, sizeof(int32_t) * part_size, hipMemcpyHostToDevice);

    int64_t *B0_part;
    int64_t *B0_idx;
    hipMalloc(&B0_idx, sizeof(int64_t));
    hipMemset(B0_idx, 0, sizeof(int64_t));
    pipeline_1<<<std::ceil((float)part_size / (float)32), 32>>>(d_p_name, d_p_name_offsets, d_p_name_sizes, d_p_partkey, d_l_partkey, part_size, B0_idx);

    int64_t h_B0_idx;
    hipMemcpy(&h_B0_idx, B0_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B0_idx, 0, sizeof(int64_t));
    hipMalloc(&B0_part, sizeof(int64_t) * h_B0_idx);
    auto HT0 = cuco::static_map{h_B0_idx * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT0_F = HT0.ref(cuco::find);

    auto d_HT0_I = HT0.ref(cuco::insert);

    pipeline_0<<<std::ceil((float)part_size / (float)32), 32>>>(d_p_name, d_p_name_offsets, d_p_name_sizes, d_l_partkey, d_p_partkey, d_HT0_I, d_HT0_F, part_size, B0_idx, B0_part);

    int32_t *d_l_orderkey;

    hipMalloc(&d_l_orderkey, sizeof(int32_t) * lineitem_size);

    hipMemcpy(d_l_orderkey, l_orderkey, sizeof(int32_t) * lineitem_size, hipMemcpyHostToDevice);

    int64_t *B1_part;
    int64_t *B1_lineitem;
    int64_t *B1_idx;
    hipMalloc(&B1_idx, sizeof(int64_t));
    hipMemset(B1_idx, 0, sizeof(int64_t));
    pipeline_3<<<std::ceil((float)lineitem_size / (float)32), 32>>>(d_l_partkey, d_l_orderkey, d_HT0_I, d_HT0_F, lineitem_size, B0_part, B1_idx);

    int64_t h_B1_idx;
    hipMemcpy(&h_B1_idx, B1_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B1_idx, 0, sizeof(int64_t));
    hipMalloc(&B1_part, sizeof(int64_t) * h_B1_idx);
    hipMalloc(&B1_lineitem, sizeof(int64_t) * h_B1_idx);
    auto HT1 = cuco::experimental::static_multimap{h_B1_idx * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, {}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>(), {}, cuco::storage<2>{}};

    auto d_HT1_F = HT1.ref(cuco::for_each);

    auto d_HT1_I = HT1.ref(cuco::insert);

    pipeline_2<<<std::ceil((float)lineitem_size / (float)32), 32>>>(d_l_partkey, d_l_orderkey, d_HT0_I, d_HT0_F, d_HT1_I, d_HT1_F, lineitem_size, B1_lineitem, B1_part, B0_part, B1_idx);

    int32_t *d_o_orderkey;

    hipMalloc(&d_o_orderkey, sizeof(int32_t) * orders_size);

    hipMemcpy(d_o_orderkey, o_orderkey, sizeof(int32_t) * orders_size, hipMemcpyHostToDevice);

    int32_t *d_l_suppkey;

    hipMalloc(&d_l_suppkey, sizeof(int32_t) * lineitem_size);

    hipMemcpy(d_l_suppkey, l_suppkey, sizeof(int32_t) * lineitem_size, hipMemcpyHostToDevice);

    int64_t *B4_orders;
    int64_t *B4_part;
    int64_t *B4_lineitem;
    int64_t *B4_idx;
    hipMalloc(&B4_idx, sizeof(int64_t));
    hipMemset(B4_idx, 0, sizeof(int64_t));
    pipeline_5<<<std::ceil((float)orders_size / (float)32), 32>>>(d_l_partkey, d_o_orderkey, d_l_suppkey, d_HT1_I, d_HT1_F, B1_part, orders_size, B4_idx, B1_lineitem);

    int64_t h_B4_idx;
    hipMemcpy(&h_B4_idx, B4_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B4_idx, 0, sizeof(int64_t));
    hipMalloc(&B4_orders, sizeof(int64_t) * h_B4_idx);
    hipMalloc(&B4_part, sizeof(int64_t) * h_B4_idx);
    hipMalloc(&B4_lineitem, sizeof(int64_t) * h_B4_idx);
    auto HT4 = cuco::experimental::static_multimap{h_B4_idx * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, {}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>(), {}, cuco::storage<2>{}};

    auto d_HT4_F = HT4.ref(cuco::for_each);

    auto d_HT4_I = HT4.ref(cuco::insert);

    pipeline_4<<<std::ceil((float)orders_size / (float)32), 32>>>(d_l_partkey, d_o_orderkey, d_l_suppkey, d_HT1_I, d_HT1_F, d_HT4_I, d_HT4_F, B4_idx, B4_orders, B1_lineitem, B1_part, orders_size, B4_lineitem, B4_part);

    int32_t *d_n_nationkey;

    hipMalloc(&d_n_nationkey, sizeof(int32_t) * nation_size);

    hipMemcpy(d_n_nationkey, n_nationkey, sizeof(int32_t) * nation_size, hipMemcpyHostToDevice);

    int32_t *d_s_nationkey;

    hipMalloc(&d_s_nationkey, sizeof(int32_t) * supplier_size);

    hipMemcpy(d_s_nationkey, s_nationkey, sizeof(int32_t) * supplier_size, hipMemcpyHostToDevice);

    int64_t *B2_nation;
    int64_t *B2_idx;
    hipMalloc(&B2_idx, sizeof(int64_t));
    hipMemset(B2_idx, 0, sizeof(int64_t));
    pipeline_7<<<std::ceil((float)nation_size / (float)32), 32>>>(d_n_nationkey, d_s_nationkey, B2_idx, nation_size);

    int64_t h_B2_idx;
    hipMemcpy(&h_B2_idx, B2_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B2_idx, 0, sizeof(int64_t));
    hipMalloc(&B2_nation, sizeof(int64_t) * h_B2_idx);
    auto HT2 = cuco::static_map{h_B2_idx * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT2_F = HT2.ref(cuco::find);

    auto d_HT2_I = HT2.ref(cuco::insert);

    pipeline_6<<<std::ceil((float)nation_size / (float)32), 32>>>(d_n_nationkey, d_s_nationkey, d_HT2_I, d_HT2_F, B2_idx, B2_nation, nation_size);

    int32_t *d_ps_suppkey;

    hipMalloc(&d_ps_suppkey, sizeof(int32_t) * partsupp_size);

    hipMemcpy(d_ps_suppkey, ps_suppkey, sizeof(int32_t) * partsupp_size, hipMemcpyHostToDevice);

    int32_t *d_s_suppkey;

    hipMalloc(&d_s_suppkey, sizeof(int32_t) * supplier_size);

    hipMemcpy(d_s_suppkey, s_suppkey, sizeof(int32_t) * supplier_size, hipMemcpyHostToDevice);

    int64_t *B3_supplier;
    int64_t *B3_nation;
    int64_t *B3_idx;
    hipMalloc(&B3_idx, sizeof(int64_t));
    hipMemset(B3_idx, 0, sizeof(int64_t));
    pipeline_9<<<std::ceil((float)supplier_size / (float)32), 32>>>(d_s_nationkey, d_s_suppkey, d_ps_suppkey, d_HT2_I, d_HT2_F, B3_idx, supplier_size, B2_nation);

    int64_t h_B3_idx;
    hipMemcpy(&h_B3_idx, B3_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B3_idx, 0, sizeof(int64_t));
    hipMalloc(&B3_supplier, sizeof(int64_t) * h_B3_idx);
    hipMalloc(&B3_nation, sizeof(int64_t) * h_B3_idx);
    auto HT3 = cuco::static_map{h_B3_idx * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT3_F = HT3.ref(cuco::find);

    auto d_HT3_I = HT3.ref(cuco::insert);

    pipeline_8<<<std::ceil((float)supplier_size / (float)32), 32>>>(d_ps_suppkey, d_s_suppkey, d_s_nationkey, d_HT2_I, d_HT2_F, d_HT3_I, d_HT3_F, B3_nation, B3_supplier, supplier_size, B2_nation, B3_idx);

    int32_t *d_ps_partkey;

    hipMalloc(&d_ps_partkey, sizeof(int32_t) * partsupp_size);

    hipMemcpy(d_ps_partkey, ps_partkey, sizeof(int32_t) * partsupp_size, hipMemcpyHostToDevice);

    double *d_l_extendedprice;

    hipMalloc(&d_l_extendedprice, sizeof(double) * lineitem_size);

    hipMemcpy(d_l_extendedprice, l_extendedprice, sizeof(double) * lineitem_size, hipMemcpyHostToDevice);

    double *d_ps_supplycost;

    hipMalloc(&d_ps_supplycost, sizeof(double) * partsupp_size);

    hipMemcpy(d_ps_supplycost, ps_supplycost, sizeof(double) * partsupp_size, hipMemcpyHostToDevice);

    double *d_l_discount;

    hipMalloc(&d_l_discount, sizeof(double) * lineitem_size);

    hipMemcpy(d_l_discount, l_discount, sizeof(double) * lineitem_size, hipMemcpyHostToDevice);

    int8_t *d_n_name;

    hipMalloc(&d_n_name, sizeof(int8_t) * nation_size);

    hipMemcpy(d_n_name, n_name, sizeof(int8_t) * nation_size, hipMemcpyHostToDevice);

    int64_t *d_l_quantity;

    hipMalloc(&d_l_quantity, sizeof(int64_t) * lineitem_size);

    hipMemcpy(d_l_quantity, l_quantity, sizeof(int64_t) * lineitem_size, hipMemcpyHostToDevice);

    int32_t *d_o_orderdate;

    hipMalloc(&d_o_orderdate, sizeof(int32_t) * orders_size);

    hipMemcpy(d_o_orderdate, o_orderdate, sizeof(int32_t) * orders_size, hipMemcpyHostToDevice);

    int32_t *d_agg_o_orderdate;

    double *d_sum_profit;

    int8_t *d_agg_n_name;

    auto HT5 = cuco::static_map{partsupp_size * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT5_F = HT5.ref(cuco::find);

    auto d_HT5_I = HT5.ref(cuco::insert);

    pipeline_10<<<std::ceil((float)partsupp_size / (float)32), 32>>>(d_ps_partkey, d_ps_suppkey, d_l_extendedprice, d_l_discount, d_ps_supplycost, d_n_name, d_s_suppkey, d_l_quantity, d_o_orderdate, d_HT3_I, d_HT3_F, d_HT4_I, d_HT4_F, d_HT5_I, d_HT5_F, B3_nation, B3_supplier, B4_orders, B4_lineitem, partsupp_size, B4_part);

    auto HT5_size = HT5.size();

    hipMalloc(&d_agg_o_orderdate, sizeof(int32_t) * HT5_size);

    hipMalloc(&d_sum_profit, sizeof(double) * HT5_size);

    hipMalloc(&d_agg_n_name, sizeof(int8_t) * HT5_size);

    hipMemset(d_agg_o_orderdate, 0, sizeof(int32_t) * HT5_size);

    hipMemset(d_sum_profit, 0, sizeof(double) * HT5_size);

    hipMemset(d_agg_n_name, 0, sizeof(int8_t) * HT5_size);

    thrust::device_vector<int64_t> keys_5(HT5_size), vals_5(HT5_size);
    HT5.retrieve_all(keys_5.begin(), vals_5.begin());
    thrust::host_vector<int64_t> h_keys_5(HT5_size);
    thrust::copy(keys_5.begin(), keys_5.end(), h_keys_5.begin());
    thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_5(HT5_size);
    for (int i = 0; i < HT5_size; i++)
    {
        actual_dict_5[i] = cuco::make_pair(h_keys_5[i], i);
    }
    HT5.clear();
    HT5.insert(actual_dict_5.begin(), actual_dict_5.end());
    pipeline_11<<<std::ceil((float)partsupp_size / (float)32), 32>>>(d_ps_partkey, d_ps_suppkey, d_l_extendedprice, d_ps_supplycost, d_l_discount, d_n_name, d_s_suppkey, d_l_quantity, d_o_orderdate, d_agg_o_orderdate, d_sum_profit, d_agg_n_name, d_HT3_I, d_HT3_F, d_HT4_I, d_HT4_F, d_HT5_I, d_HT5_F, B3_nation, B3_supplier, B4_orders, B4_lineitem, partsupp_size, B4_part);

    size_t agg_size = HT5_size;
    int8_t *p_agg_n_name = (int8_t *)malloc(sizeof(int8_t) * agg_size);
    hipMemcpy(p_agg_n_name, d_agg_n_name, sizeof(int8_t) * agg_size, hipMemcpyDeviceToHost);
    int32_t *p_agg_o_orderdate = (int32_t *)malloc(sizeof(int32_t) * agg_size);
    hipMemcpy(p_agg_o_orderdate, d_agg_o_orderdate, sizeof(int32_t) * agg_size, hipMemcpyDeviceToHost);
    double *p_sum_profit = (double *)malloc(sizeof(double) * agg_size);
    hipMemcpy(p_sum_profit, d_sum_profit, sizeof(double) * agg_size, hipMemcpyDeviceToHost);
    for (int i = 0; i < agg_size; i++)
    {
        std::cout << (int)p_agg_n_name[i] << "\t";
        std::cout << p_agg_o_orderdate[i] << "\t";
        std::cout << p_sum_profit[i] << "\t";
        std::cout << std::endl;
    }
}

int main(int argc, const char **argv)
{
    std::string dbDir = getDataDir(argv, argc);
    std::string lineitem_file = dbDir + "lineitem.parquet";
    std::string nation_file = dbDir + "nation.parquet";
    std::string orders_file = dbDir + "orders.parquet";
    std::string part_file = dbDir + "part.parquet";
    std::string partsupp_file = dbDir + "partsupp.parquet";
    std::string supplier_file = dbDir + "supplier.parquet";

    // auto lineitem_table  = getArrowTable(lineitem_file);
    auto lineitem_table = getArrowTable(lineitem_file);
    auto nation_table = getArrowTable(nation_file);
    auto orders_table = getArrowTable(orders_file);
    auto part_table = getArrowTable(part_file);
    auto partsupp_table = getArrowTable(partsupp_file);
    auto supplier_table = getArrowTable(supplier_file);
    size_t lineitem_size = lineitem_table->num_rows();
    size_t nation_size = nation_table->num_rows();
    size_t orders_size = orders_table->num_rows();
    size_t part_size = part_table->num_rows();
    size_t partsupp_size = partsupp_table->num_rows();
    size_t supplier_size = supplier_table->num_rows();

    auto l_extendedprice = read_column<double>(lineitem_table, "l_extendedprice");
    auto l_discount = read_column<double>(lineitem_table, "l_discount");
    auto l_orderkey = read_column_typecasted<int32_t>(lineitem_table, "l_orderkey");
    auto l_partkey = read_column_typecasted<int32_t>(lineitem_table, "l_partkey");
    auto l_quantity = read_column<int64_t>(lineitem_table, "l_quantity");
    auto l_suppkey = read_column_typecasted<int32_t>(lineitem_table, "l_suppkey");

    auto n_name = read_string_dict_encoded_column(nation_table, "n_name");
    auto n_nationkey = read_column_typecasted<int32_t>(nation_table, "n_nationkey");

    auto o_orderkey = read_column_typecasted<int32_t>(orders_table, "o_orderkey");
    auto o_orderdate = read_column<int32_t>(orders_table, "o_orderdate");

    auto p_name = read_string_column(part_table, "p_name");
    auto p_partkey = read_column_typecasted<int32_t>(part_table, "p_partkey");

    auto ps_partkey = read_column_typecasted<int32_t>(partsupp_table, "ps_partkey");
    auto ps_suppkey = read_column_typecasted<int32_t>(partsupp_table, "ps_suppkey");
    auto ps_supplycost = read_column<double>(partsupp_table, "ps_supplycost");

    auto s_nationkey = read_column_typecasted<int32_t>(supplier_table, "s_nationkey");
    auto s_suppkey = read_column_typecasted<int32_t>(supplier_table, "s_suppkey");

    // for (auto p: c_mktsegment->dict) {
    //   std::cout << p.first << " " << (int)p.second << std::endl;
    // }
    control(
        l_discount.data(),
        l_extendedprice.data(),
        l_orderkey.data(),
        l_partkey.data(),
        l_quantity.data(),
        l_suppkey.data(),
        n_name->column,
        n_nationkey.data(),
        o_orderdate.data(),
        o_orderkey.data(),
        p_name,
        p_partkey.data(),
        ps_partkey.data(),
        ps_suppkey.data(),
        ps_supplycost.data(),
        s_nationkey.data(),
        s_suppkey.data(),
        lineitem_size,
        nation_size,
        orders_size,
        part_size,
        partsupp_size,
        supplier_size);
}