#include "hip/hip_runtime.h"
#include "utils.h"

#include <cuco/static_map.cuh>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace cg = cooperative_groups;
template <typename TY_HT0_I, typename TY_HT0_F>
__global__ void pipeline_0 (int32_t* l_orderkey, TY_HT0_I HT0_I, TY_HT0_F HT0_F, size_t lineitem_size) {
int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid >= lineitem_size) return;
int32_t reg_l_orderkey = l_orderkey[tid];
int64_t key0 = 0;
key0 |= (((int64_t)reg_l_orderkey) << 0);
auto thread = cg::tiled_partition<1>(cg::this_thread_block());
HT0_I.insert(thread, cuco::pair{key0, 1});
}

template <typename TY_HT0_I, typename TY_HT0_F>
__global__ void pipeline_1 (int64_t* l_quantity, int32_t* l_orderkey, int64_t* agg1_l_quantity, int32_t* agg1_l_orderkey, TY_HT0_I HT0_I, TY_HT0_F HT0_F, size_t lineitem_size) {
int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid >= lineitem_size) return;
int32_t reg_l_orderkey = l_orderkey[tid];
int64_t key0 = 0;
key0 |= (((int64_t)reg_l_orderkey) << 0);
auto slot0 = HT0_F.find(key0);
int64_t reg_l_quantity = l_quantity[tid];
agg1_l_orderkey[slot0->second] = reg_l_orderkey;
aggregate_sum(&(agg1_l_quantity[slot0->second]), reg_l_quantity);
}

__global__ void pipeline_3 (int32_t* c_custkey, size_t customer_size, int64_t* B2_idx) {
int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid >= customer_size) return;
int32_t reg_c_custkey = c_custkey[tid];
int64_t key2 = 0;
key2 |= (((int64_t)reg_c_custkey) << 0);
atomicAdd((int*)B2_idx, 1);
}

template <typename TY_HT2_I, typename TY_HT2_F>
__global__ void pipeline_2 (int32_t* c_custkey, TY_HT2_I HT2_I, TY_HT2_F HT2_F, size_t customer_size, int64_t* B2_customer, int64_t* B2_idx) {
int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid >= customer_size) return;
int32_t reg_c_custkey = c_custkey[tid];
int64_t key2 = 0;
key2 |= (((int64_t)reg_c_custkey) << 0);
auto reg_B2_idx = atomicAdd((int*)B2_idx, 1);
auto thread = cg::tiled_partition<1>(cg::this_thread_block());
HT2_I.insert(thread, cuco::pair{key2, reg_B2_idx});
B2_customer[reg_B2_idx] = tid;
}


__global__ void pipeline_5 (int32_t* o_orderkey, int64_t* B1_idx, size_t orders_size) {
int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid >= orders_size) return;
int32_t reg_o_orderkey = o_orderkey[tid];
int64_t key1 = 0;
key1 |= (((int64_t)reg_o_orderkey) << 0);
atomicAdd((int*)B1_idx, 1);
}

template <typename TY_HT1_I, typename TY_HT1_F>
__global__ void pipeline_4 (int32_t* o_orderkey, TY_HT1_I HT1_I, TY_HT1_F HT1_F, int64_t* B1_idx, size_t orders_size, int64_t* B1_orders) {
int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid >= orders_size) return;
int32_t reg_o_orderkey = o_orderkey[tid];
int64_t key1 = 0;
key1 |= (((int64_t)reg_o_orderkey) << 0);
auto reg_B1_idx = atomicAdd((int*)B1_idx, 1);
auto thread = cg::tiled_partition<1>(cg::this_thread_block());
HT1_I.insert(thread, cuco::pair{key1, reg_B1_idx});
B1_orders[reg_B1_idx] = tid;
}

template <typename TY_HT1_I, typename TY_HT1_F, typename TY_HT2_I, typename TY_HT2_F>
__global__ void pipeline_7 (int32_t* o_orderkey, int32_t* o_custkey, int64_t* agg1_l_quantity, int32_t* agg1_l_orderkey, TY_HT1_I HT1_I, TY_HT1_F HT1_F, TY_HT2_I HT2_I, TY_HT2_F HT2_F, int64_t* B2_customer, int64_t* B1_orders, int64_t* B3_idx, size_t agg1_size) {
int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid >= agg1_size) return;
int64_t reg_agg1_l_quantity = agg1_l_quantity[tid];
if (!(reg_agg1_l_quantity  > 300)) return;
int32_t reg_agg1_l_orderkey = agg1_l_orderkey[tid];
int64_t key1 = 0;
key1 |= (((int64_t)reg_agg1_l_orderkey) << 0);
auto slot1 = HT1_F.find(key1);
if (slot1 == HT1_F.end()) return;int32_t reg_o_custkey = o_custkey[B1_orders[slot1->second]];
int64_t key2 = 0;
key2 |= (((int64_t)reg_o_custkey) << 0);
auto slot2 = HT2_F.find(key2);
if (slot2 == HT2_F.end()) return;int32_t reg_o_orderkey = o_orderkey[B1_orders[slot1->second]];
int64_t key3 = 0;
key3 |= (((int64_t)reg_o_orderkey) << 0);
atomicAdd((int*)B3_idx, 1);
}

template <typename TY_HT1_I, typename TY_HT1_F, typename TY_HT2_I, typename TY_HT2_F, typename TY_HT3_I, typename TY_HT3_F>
__global__ void pipeline_6 (int32_t* o_orderkey, int32_t* o_custkey, int64_t* agg1_l_quantity, int32_t* agg1_l_orderkey, TY_HT1_I HT1_I, TY_HT1_F HT1_F, TY_HT2_I HT2_I, TY_HT2_F HT2_F, TY_HT3_I HT3_I, TY_HT3_F HT3_F, int64_t* B1_orders, int64_t* B3_agg1, int64_t* B3_idx, size_t agg1_size, int64_t* B2_customer, int64_t* B3_orders, int64_t* B3_customer) {
int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid >= agg1_size) return;
int64_t reg_agg1_l_quantity = agg1_l_quantity[tid];
if (!(reg_agg1_l_quantity  > 300)) return;
int32_t reg_agg1_l_orderkey = agg1_l_orderkey[tid];
int64_t key1 = 0;
key1 |= (((int64_t)reg_agg1_l_orderkey) << 0);
auto slot1 = HT1_F.find(key1);
if (slot1 == HT1_F.end()) return;int32_t reg_o_custkey = o_custkey[B1_orders[slot1->second]];
int64_t key2 = 0;
key2 |= (((int64_t)reg_o_custkey) << 0);
auto slot2 = HT2_F.find(key2);
if (slot2 == HT2_F.end()) return;int32_t reg_o_orderkey = o_orderkey[B1_orders[slot1->second]];
int64_t key3 = 0;
key3 |= (((int64_t)reg_o_orderkey) << 0);
auto reg_B3_idx = atomicAdd((int*)B3_idx, 1);
auto thread = cg::tiled_partition<1>(cg::this_thread_block());
HT3_I.insert(thread, cuco::pair{key3, reg_B3_idx});
B3_orders[reg_B3_idx] = B1_orders[slot1->second];
B3_agg1[reg_B3_idx] = tid;
B3_customer[reg_B3_idx] = B2_customer[slot2->second];
}

template <typename TY_HT3_I, typename TY_HT3_F, typename TY_HT4_I, typename TY_HT4_F>
__global__ void pipeline_8 (int32_t* l_orderkey, int32_t* o_orderkey, TY_HT3_I HT3_I, TY_HT3_F HT3_F, TY_HT4_I HT4_I, TY_HT4_F HT4_F, int64_t* B3_customer, size_t lineitem_size, int64_t* B3_orders, int64_t* B3_agg1) {
int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid >= lineitem_size) return;
int32_t reg_l_orderkey = l_orderkey[tid];
int64_t key3 = 0;
key3 |= (((int64_t)reg_l_orderkey) << 0);
auto slot3 = HT3_F.find(key3);
if (slot3 == HT3_F.end()) return;int32_t reg_o_orderkey = o_orderkey[B3_orders[slot3->second]];
int64_t key4 = 0;
key4 |= (((int64_t)reg_o_orderkey) << 0);
auto thread = cg::tiled_partition<1>(cg::this_thread_block());
HT4_I.insert(thread, cuco::pair{key4, 1});
}

template <typename TY_HT3_I, typename TY_HT3_F, typename TY_HT4_I, typename TY_HT4_F>
__global__ void pipeline_9 (int64_t* l_quantity, int32_t* l_orderkey, int32_t* o_orderkey, int32_t* agg2_o_orderkey, int64_t* agg2_l_quantity, TY_HT3_I HT3_I, TY_HT3_F HT3_F, TY_HT4_I HT4_I, TY_HT4_F HT4_F, int64_t* B3_agg1, size_t lineitem_size, int64_t* B3_orders, int64_t* B3_customer) {
int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
if (tid >= lineitem_size) return;
int32_t reg_l_orderkey = l_orderkey[tid];
int64_t key3 = 0;
key3 |= (((int64_t)reg_l_orderkey) << 0);
auto slot3 = HT3_F.find(key3);
if (slot3 == HT3_F.end()) return;int32_t reg_o_orderkey = o_orderkey[B3_orders[slot3->second]];
int64_t key4 = 0;
key4 |= (((int64_t)reg_o_orderkey) << 0);
auto slot4 = HT4_F.find(key4);
int64_t reg_l_quantity = l_quantity[tid];
agg2_o_orderkey[slot4->second] = reg_o_orderkey;
aggregate_sum(&(agg2_l_quantity[slot4->second]), reg_l_quantity);
}

void control(
int32_t *c_custkey,
int32_t *l_orderkey,
int64_t *l_quantity,
int32_t *o_custkey,
int32_t *o_orderkey,
size_t customer_size,
size_t lineitem_size,
size_t orders_size
) {
    int64_t* d_l_quantity;

hipMalloc(&d_l_quantity, sizeof(int64_t) * lineitem_size);

hipMemcpy(d_l_quantity, l_quantity, sizeof(int64_t) * lineitem_size, hipMemcpyHostToDevice);

int32_t* d_l_orderkey;

hipMalloc(&d_l_orderkey, sizeof(int32_t) * lineitem_size);

hipMemcpy(d_l_orderkey, l_orderkey, sizeof(int32_t) * lineitem_size, hipMemcpyHostToDevice);

int64_t* d_agg1_l_quantity;

int32_t* d_agg1_l_orderkey;

auto HT0 = cuco::static_map{ lineitem_size * 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

auto d_HT0_F = HT0.ref(cuco::find);

auto d_HT0_I = HT0.ref(cuco::insert);

pipeline_0<<<std::ceil((float)lineitem_size/(float)32), 32>>>(d_l_orderkey, d_HT0_I, d_HT0_F, lineitem_size);

auto HT0_size = HT0.size();

hipMalloc(&d_agg1_l_quantity, sizeof(int64_t) * HT0_size);

hipMalloc(&d_agg1_l_orderkey, sizeof(int32_t) * HT0_size);

hipMemset(d_agg1_l_quantity, 0, sizeof(int64_t) * HT0_size);

hipMemset(d_agg1_l_orderkey, 0, sizeof(int32_t) * HT0_size);

thrust::device_vector<int64_t> keys_0(HT0_size), vals_0(HT0_size);
HT0.retrieve_all(keys_0.begin(), vals_0.begin());
thrust::host_vector<int64_t> h_keys_0(HT0_size);
thrust::copy(keys_0.begin(), keys_0.end(), h_keys_0.begin());
thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_0(HT0_size);
for (int i=0; i < HT0_size; i++) {
actual_dict_0[i] = cuco::make_pair(h_keys_0[i], i);
}
HT0.clear();
HT0.insert(actual_dict_0.begin(), actual_dict_0.end());
pipeline_1<<<std::ceil((float)lineitem_size/(float)32), 32>>>(d_l_quantity, d_l_orderkey, d_agg1_l_quantity, d_agg1_l_orderkey, d_HT0_I, d_HT0_F, lineitem_size);

size_t agg1_size = HT0_size;
int32_t* d_c_custkey;

hipMalloc(&d_c_custkey, sizeof(int32_t) * customer_size);

hipMemcpy(d_c_custkey, c_custkey, sizeof(int32_t) * customer_size, hipMemcpyHostToDevice);

int64_t* B2_customer;
int64_t* B2_idx;
hipMalloc(&B2_idx, sizeof(int64_t));
hipMemset(B2_idx, 0, sizeof(int64_t));
pipeline_3<<<std::ceil((float)customer_size/(float)32), 32>>>(d_c_custkey, customer_size, B2_idx);

int64_t h_B2_idx;
hipMemcpy(&h_B2_idx, B2_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
hipMemset(B2_idx, 0, sizeof(int64_t));
hipMalloc(&B2_customer, sizeof(int64_t) * h_B2_idx);
auto HT2 = cuco::static_map{ h_B2_idx * 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

auto d_HT2_F = HT2.ref(cuco::find);

auto d_HT2_I = HT2.ref(cuco::insert);

pipeline_2<<<std::ceil((float)customer_size/(float)32), 32>>>(d_c_custkey, d_HT2_I, d_HT2_F, customer_size, B2_customer, B2_idx);

int32_t* d_o_orderkey;

hipMalloc(&d_o_orderkey, sizeof(int32_t) * orders_size);

hipMemcpy(d_o_orderkey, o_orderkey, sizeof(int32_t) * orders_size, hipMemcpyHostToDevice);

int64_t* B1_orders;
int64_t* B1_idx;
hipMalloc(&B1_idx, sizeof(int64_t));
hipMemset(B1_idx, 0, sizeof(int64_t));
pipeline_5<<<std::ceil((float)orders_size/(float)32), 32>>>(d_o_orderkey, B1_idx, orders_size);

int64_t h_B1_idx;
hipMemcpy(&h_B1_idx, B1_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
hipMemset(B1_idx, 0, sizeof(int64_t));
hipMalloc(&B1_orders, sizeof(int64_t) * h_B1_idx);
auto HT1 = cuco::static_map{ h_B1_idx * 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

auto d_HT1_F = HT1.ref(cuco::find);

auto d_HT1_I = HT1.ref(cuco::insert);

pipeline_4<<<std::ceil((float)orders_size/(float)32), 32>>>(d_o_orderkey, d_HT1_I, d_HT1_F, B1_idx, orders_size, B1_orders);

int32_t* d_o_custkey;

hipMalloc(&d_o_custkey, sizeof(int32_t) * orders_size);

hipMemcpy(d_o_custkey, o_custkey, sizeof(int32_t) * orders_size, hipMemcpyHostToDevice);

int64_t* B3_orders;
int64_t* B3_agg1;
int64_t* B3_customer;
int64_t* B3_idx;
hipMalloc(&B3_idx, sizeof(int64_t));
hipMemset(B3_idx, 0, sizeof(int64_t));
pipeline_7<<<std::ceil((float)agg1_size/(float)32), 32>>>(d_o_orderkey, d_o_custkey, d_agg1_l_quantity, d_agg1_l_orderkey, d_HT1_I, d_HT1_F, d_HT2_I, d_HT2_F, B2_customer, B1_orders, B3_idx, agg1_size);

int64_t h_B3_idx;
hipMemcpy(&h_B3_idx, B3_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
hipMemset(B3_idx, 0, sizeof(int64_t));
hipMalloc(&B3_orders, sizeof(int64_t) * h_B3_idx);
hipMalloc(&B3_agg1, sizeof(int64_t) * h_B3_idx);
hipMalloc(&B3_customer, sizeof(int64_t) * h_B3_idx);
auto HT3 = cuco::static_map{ h_B3_idx * 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

auto d_HT3_F = HT3.ref(cuco::find);

auto d_HT3_I = HT3.ref(cuco::insert);

pipeline_6<<<std::ceil((float)agg1_size/(float)32), 32>>>(d_o_orderkey, d_o_custkey, d_agg1_l_quantity, d_agg1_l_orderkey, d_HT1_I, d_HT1_F, d_HT2_I, d_HT2_F, d_HT3_I, d_HT3_F, B1_orders, B3_agg1, B3_idx, agg1_size, B2_customer, B3_orders, B3_customer);

int32_t* d_agg2_o_orderkey;

int64_t* d_agg2_l_quantity;

auto HT4 = cuco::static_map{ lineitem_size * 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

auto d_HT4_F = HT4.ref(cuco::find);

auto d_HT4_I = HT4.ref(cuco::insert);

pipeline_8<<<std::ceil((float)lineitem_size/(float)32), 32>>>(d_l_orderkey, d_o_orderkey, d_HT3_I, d_HT3_F, d_HT4_I, d_HT4_F, B3_customer, lineitem_size, B3_orders, B3_agg1);

auto HT4_size = HT4.size();

hipMalloc(&d_agg2_o_orderkey, sizeof(int32_t) * HT4_size);

hipMalloc(&d_agg2_l_quantity, sizeof(int64_t) * HT4_size);

hipMemset(d_agg2_o_orderkey, 0, sizeof(int32_t) * HT4_size);

hipMemset(d_agg2_l_quantity, 0, sizeof(int64_t) * HT4_size);

thrust::device_vector<int64_t> keys_4(HT4_size), vals_4(HT4_size);
HT4.retrieve_all(keys_4.begin(), vals_4.begin());
thrust::host_vector<int64_t> h_keys_4(HT4_size);
thrust::copy(keys_4.begin(), keys_4.end(), h_keys_4.begin());
thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_4(HT4_size);
for (int i=0; i < HT4_size; i++) {
actual_dict_4[i] = cuco::make_pair(h_keys_4[i], i);
}
HT4.clear();
HT4.insert(actual_dict_4.begin(), actual_dict_4.end());
pipeline_9<<<std::ceil((float)lineitem_size/(float)32), 32>>>(d_l_quantity, d_l_orderkey, d_o_orderkey, d_agg2_o_orderkey, d_agg2_l_quantity, d_HT3_I, d_HT3_F, d_HT4_I, d_HT4_F, B3_agg1, lineitem_size, B3_orders, B3_customer);

size_t agg2_size = HT4_size;
int32_t* p_agg2_o_orderkey = (int32_t*)malloc(sizeof(int32_t) * agg2_size);
hipMemcpy(p_agg2_o_orderkey, d_agg2_o_orderkey, sizeof(int32_t) * agg2_size, hipMemcpyDeviceToHost);
int64_t* p_agg2_l_quantity = (int64_t*)malloc(sizeof(int64_t) * agg2_size);
hipMemcpy(p_agg2_l_quantity, d_agg2_l_quantity, sizeof(int64_t) * agg2_size, hipMemcpyDeviceToHost);
for (int i=0; i<agg2_size; i++) {
std::cout << p_agg2_o_orderkey[i] << "\t";
std::cout << p_agg2_l_quantity[i] << "\t";
std::cout << std::endl;
}
}

int main(int argc, const char** argv)
{
  std::string dbDir         = getDataDir(argv, argc);
  std::string lineitem_file = dbDir + "lineitem.parquet";
  std::string customer_file = dbDir + "customer.parquet";
  std::string orders_file   = dbDir + "orders.parquet";

  auto lineitem_table  = getArrowTable(lineitem_file);
  auto customer_table  = getArrowTable(customer_file);
  auto orders_table    = getArrowTable(orders_file);
  size_t lineitem_size = lineitem_table->num_rows();
  size_t customer_size = customer_table->num_rows();
  size_t orders_size   = orders_table->num_rows();

  auto l_orderkey = read_column_typecasted<int32_t>(lineitem_table, "l_orderkey");
  auto l_quantity = read_column<int64_t>(lineitem_table, "l_quantity");

  auto c_custkey = read_column_typecasted<int32_t>(customer_table, "c_custkey");

  auto o_custkey  = read_column_typecasted<int32_t>(orders_table, "o_custkey");
  auto o_orderkey = read_column_typecasted<int32_t>(orders_table, "o_orderkey");

  // for (auto p: c_mktsegment->dict) {
  //   std::cout << p.first << " " << (int)p.second << std::endl;
  // }
  control(c_custkey.data(),
          l_orderkey.data(),
          l_quantity.data(),
          o_custkey.data(),
          o_orderkey.data(),
          customer_size,
          lineitem_size,
          orders_size);
}