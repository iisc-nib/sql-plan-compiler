#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_58453863ae50(uint64_t* COUNT58453862c730, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
//Materialize count
atomicAdd((int*)COUNT58453862c730, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_58453863ae50(uint64_t* BUF_58453862c730, uint64_t* BUF_IDX_58453862c730, HASHTABLE_INSERT HT_58453862c730, DBI32Type* nation__n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_58453862c730 = 0;
auto reg_nation__n_nationkey = nation__n_nationkey[tid];

KEY_58453862c730 |= reg_nation__n_nationkey;
// Insert hash table kernel;
auto buf_idx_58453862c730 = atomicAdd((int*)BUF_IDX_58453862c730, 1);
HT_58453862c730.insert(cuco::pair{KEY_58453862c730, buf_idx_58453862c730});
BUF_58453862c730[buf_idx_58453862c730 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5845385b6ea0(uint64_t* BUF_58453862c730, uint64_t* COUNT58453862ccc0, HASHTABLE_PROBE HT_58453862c730, DBI32Type* supplier__s_nationkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_58453862c730 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_58453862c730 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_58453862c730.for_each(KEY_58453862c730, [&] __device__ (auto const SLOT_58453862c730) {

auto const [slot_first58453862c730, slot_second58453862c730] = SLOT_58453862c730;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT58453862ccc0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5845385b6ea0(uint64_t* BUF_58453862c730, uint64_t* BUF_58453862ccc0, uint64_t* BUF_IDX_58453862ccc0, HASHTABLE_PROBE HT_58453862c730, HASHTABLE_INSERT HT_58453862ccc0, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_58453862c730 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_58453862c730 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_58453862c730.for_each(KEY_58453862c730, [&] __device__ (auto const SLOT_58453862c730) {
auto const [slot_first58453862c730, slot_second58453862c730] = SLOT_58453862c730;
if (!(true)) return;
uint64_t KEY_58453862ccc0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_58453862ccc0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_58453862ccc0 = atomicAdd((int*)BUF_IDX_58453862ccc0, 1);
HT_58453862ccc0.insert(cuco::pair{KEY_58453862ccc0, buf_idx_58453862ccc0});
BUF_58453862ccc0[buf_idx_58453862ccc0 * 2 + 0] = tid;
BUF_58453862ccc0[buf_idx_58453862ccc0 * 2 + 1] = BUF_58453862c730[slot_second58453862c730 * 1 + 0];
});
}
__global__ void count_5845386402c0(uint64_t* COUNT584538630340, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
//Materialize count
atomicAdd((int*)COUNT584538630340, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5845386402c0(uint64_t* BUF_584538630340, uint64_t* BUF_IDX_584538630340, HASHTABLE_INSERT HT_584538630340, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_584538630340 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_584538630340 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_584538630340 = atomicAdd((int*)BUF_IDX_584538630340, 1);
HT_584538630340.insert(cuco::pair{KEY_584538630340, buf_idx_584538630340});
BUF_584538630340[buf_idx_584538630340 * 1 + 0] = tid;
}
__global__ void count_584538533fd0(uint64_t* COUNT584538630450, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
//Materialize count
atomicAdd((int*)COUNT584538630450, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_584538533fd0(uint64_t* BUF_584538630450, uint64_t* BUF_IDX_584538630450, HASHTABLE_INSERT HT_584538630450, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
uint64_t KEY_584538630450 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_584538630450 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_584538630450 = atomicAdd((int*)BUF_IDX_584538630450, 1);
HT_584538630450.insert(cuco::pair{KEY_584538630450, buf_idx_584538630450});
BUF_584538630450[buf_idx_584538630450 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_584538639f70(uint64_t* BUF_58453862ccc0, uint64_t* COUNT5845386305c0, HASHTABLE_PROBE HT_58453862ccc0, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= partsupp_size) return;
uint64_t KEY_58453862ccc0 = 0;
auto reg_partsupp__ps_suppkey = partsupp__ps_suppkey[tid];

KEY_58453862ccc0 |= reg_partsupp__ps_suppkey;
//Probe Hash table
HT_58453862ccc0.for_each(KEY_58453862ccc0, [&] __device__ (auto const SLOT_58453862ccc0) {

auto const [slot_first58453862ccc0, slot_second58453862ccc0] = SLOT_58453862ccc0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5845386305c0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_584538639f70(uint64_t* BUF_58453862ccc0, uint64_t* BUF_5845386305c0, uint64_t* BUF_IDX_5845386305c0, HASHTABLE_PROBE HT_58453862ccc0, HASHTABLE_INSERT HT_5845386305c0, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size, DBI32Type* supplier__s_suppkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= partsupp_size) return;
uint64_t KEY_58453862ccc0 = 0;
auto reg_partsupp__ps_suppkey = partsupp__ps_suppkey[tid];

KEY_58453862ccc0 |= reg_partsupp__ps_suppkey;
//Probe Hash table
HT_58453862ccc0.for_each(KEY_58453862ccc0, [&] __device__ (auto const SLOT_58453862ccc0) {
auto const [slot_first58453862ccc0, slot_second58453862ccc0] = SLOT_58453862ccc0;
if (!(true)) return;
uint64_t KEY_5845386305c0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[BUF_58453862ccc0[slot_second58453862ccc0 * 2 + 0]];

KEY_5845386305c0 |= reg_supplier__s_suppkey;
auto reg_partsupp__ps_partkey = partsupp__ps_partkey[tid];
KEY_5845386305c0 <<= 32;
KEY_5845386305c0 |= reg_partsupp__ps_partkey;
// Insert hash table kernel;
auto buf_idx_5845386305c0 = atomicAdd((int*)BUF_IDX_5845386305c0, 1);
HT_5845386305c0.insert(cuco::pair{KEY_5845386305c0, buf_idx_5845386305c0});
BUF_5845386305c0[buf_idx_5845386305c0 * 3 + 0] = BUF_58453862ccc0[slot_second58453862ccc0 * 2 + 0];
BUF_5845386305c0[buf_idx_5845386305c0 * 3 + 1] = tid;
BUF_5845386305c0[buf_idx_5845386305c0 * 3 + 2] = BUF_58453862ccc0[slot_second58453862ccc0 * 2 + 1];
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_584538637b20(uint64_t* BUF_584538630340, uint64_t* BUF_584538630450, uint64_t* BUF_5845386305c0, HASHTABLE_INSERT HT_5845385e9110, HASHTABLE_PROBE HT_584538630340, HASHTABLE_PROBE HT_584538630450, HASHTABLE_PROBE HT_5845386305c0, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBDateType* orders__o_orderdate) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_584538630340 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_584538630340 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_584538630340.for_each(KEY_584538630340, [&] __device__ (auto const SLOT_584538630340) {

auto const [slot_first584538630340, slot_second584538630340] = SLOT_584538630340;
if (!(true)) return;
uint64_t KEY_584538630450 = 0;
auto reg_lineitem__l_partkey = lineitem__l_partkey[tid];

KEY_584538630450 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_584538630450.for_each(KEY_584538630450, [&] __device__ (auto const SLOT_584538630450) {

auto const [slot_first584538630450, slot_second584538630450] = SLOT_584538630450;
if (!(true)) return;
uint64_t KEY_5845386305c0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_5845386305c0 |= reg_lineitem__l_suppkey;
KEY_5845386305c0 <<= 32;
KEY_5845386305c0 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_5845386305c0.for_each(KEY_5845386305c0, [&] __device__ (auto const SLOT_5845386305c0) {

auto const [slot_first5845386305c0, slot_second5845386305c0] = SLOT_5845386305c0;
if (!(true)) return;
uint64_t KEY_5845385e9110 = 0;
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_584538630340[slot_second584538630340 * 1 + 0]];
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_orders__o_orderdate);

KEY_5845385e9110 |= reg_map0__tmp_attr0;
//Create aggregation hash table
HT_5845385e9110.insert(cuco::pair{KEY_5845385e9110, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_584538637b20(uint64_t* BUF_584538630340, uint64_t* BUF_584538630450, uint64_t* BUF_5845386305c0, HASHTABLE_FIND HT_5845385e9110, HASHTABLE_PROBE HT_584538630340, HASHTABLE_PROBE HT_584538630450, HASHTABLE_PROBE HT_5845386305c0, DBI64Type* KEY_5845385e9110map0__tmp_attr0, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBDateType* orders__o_orderdate, DBDecimalType* partsupp__ps_supplycost) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_584538630340 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_584538630340 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_584538630340.for_each(KEY_584538630340, [&] __device__ (auto const SLOT_584538630340) {
auto const [slot_first584538630340, slot_second584538630340] = SLOT_584538630340;
if (!(true)) return;
uint64_t KEY_584538630450 = 0;
auto reg_lineitem__l_partkey = lineitem__l_partkey[tid];

KEY_584538630450 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_584538630450.for_each(KEY_584538630450, [&] __device__ (auto const SLOT_584538630450) {
auto const [slot_first584538630450, slot_second584538630450] = SLOT_584538630450;
if (!(true)) return;
uint64_t KEY_5845386305c0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_5845386305c0 |= reg_lineitem__l_suppkey;
KEY_5845386305c0 <<= 32;
KEY_5845386305c0 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_5845386305c0.for_each(KEY_5845386305c0, [&] __device__ (auto const SLOT_5845386305c0) {
auto const [slot_first5845386305c0, slot_second5845386305c0] = SLOT_5845386305c0;
if (!(true)) return;
uint64_t KEY_5845385e9110 = 0;
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_584538630340[slot_second584538630340 * 1 + 0]];
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_orders__o_orderdate);

KEY_5845385e9110 |= reg_map0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_5845385e9110 = HT_5845385e9110.find(KEY_5845385e9110)->second;
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
auto reg_partsupp__ps_supplycost = partsupp__ps_supplycost[BUF_5845386305c0[slot_second5845386305c0 * 3 + 1]];
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = ((reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount))) - ((reg_partsupp__ps_supplycost) * (reg_lineitem__l_quantity));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_5845385e9110], reg_map0__tmp_attr1);
KEY_5845385e9110map0__tmp_attr0[buf_idx_5845385e9110] = reg_map0__tmp_attr0;
});
});
});
}
__global__ void count_58453864e8c0(size_t COUNT5845385e9110, uint64_t* COUNT5845385fc300) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5845385e9110) return;
//Materialize count
atomicAdd((int*)COUNT5845385fc300, 1);
}
__global__ void main_58453864e8c0(size_t COUNT5845385e9110, DBDecimalType* MAT5845385fc300aggr0__tmp_attr2, DBI64Type* MAT5845385fc300map0__tmp_attr0, uint64_t* MAT_IDX5845385fc300, DBDecimalType* aggr0__tmp_attr2, DBI64Type* map0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5845385e9110) return;
//Materialize buffers
auto mat_idx5845385fc300 = atomicAdd((int*)MAT_IDX5845385fc300, 1);
auto reg_map0__tmp_attr0 = map0__tmp_attr0[tid];
MAT5845385fc300map0__tmp_attr0[mat_idx5845385fc300] = reg_map0__tmp_attr0;
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT5845385fc300aggr0__tmp_attr2[mat_idx5845385fc300] = reg_aggr0__tmp_attr2;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT58453862c730;
hipMalloc(&d_COUNT58453862c730, sizeof(uint64_t));
hipMemset(d_COUNT58453862c730, 0, sizeof(uint64_t));
count_58453863ae50<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT58453862c730, nation_size);
uint64_t COUNT58453862c730;
hipMemcpy(&COUNT58453862c730, d_COUNT58453862c730, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT58453862c730);
// Insert hash table control;
uint64_t* d_BUF_IDX_58453862c730;
hipMalloc(&d_BUF_IDX_58453862c730, sizeof(uint64_t));
hipMemset(d_BUF_IDX_58453862c730, 0, sizeof(uint64_t));
uint64_t* d_BUF_58453862c730;
hipMalloc(&d_BUF_58453862c730, sizeof(uint64_t) * COUNT58453862c730 * 1);
auto d_HT_58453862c730 = cuco::experimental::static_multimap{ (int)COUNT58453862c730*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_58453863ae50<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_58453862c730, d_BUF_IDX_58453862c730, d_HT_58453862c730.ref(cuco::insert), d_nation__n_nationkey, nation_size);
hipFree(d_BUF_IDX_58453862c730);
//Materialize count
uint64_t* d_COUNT58453862ccc0;
hipMalloc(&d_COUNT58453862ccc0, sizeof(uint64_t));
hipMemset(d_COUNT58453862ccc0, 0, sizeof(uint64_t));
count_5845385b6ea0<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_58453862c730, d_COUNT58453862ccc0, d_HT_58453862c730.ref(cuco::for_each), d_supplier__s_nationkey, supplier_size);
uint64_t COUNT58453862ccc0;
hipMemcpy(&COUNT58453862ccc0, d_COUNT58453862ccc0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT58453862ccc0);
// Insert hash table control;
uint64_t* d_BUF_IDX_58453862ccc0;
hipMalloc(&d_BUF_IDX_58453862ccc0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_58453862ccc0, 0, sizeof(uint64_t));
uint64_t* d_BUF_58453862ccc0;
hipMalloc(&d_BUF_58453862ccc0, sizeof(uint64_t) * COUNT58453862ccc0 * 2);
auto d_HT_58453862ccc0 = cuco::experimental::static_multimap{ (int)COUNT58453862ccc0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5845385b6ea0<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_58453862c730, d_BUF_58453862ccc0, d_BUF_IDX_58453862ccc0, d_HT_58453862c730.ref(cuco::for_each), d_HT_58453862ccc0.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
hipFree(d_BUF_IDX_58453862ccc0);
//Materialize count
uint64_t* d_COUNT584538630340;
hipMalloc(&d_COUNT584538630340, sizeof(uint64_t));
hipMemset(d_COUNT584538630340, 0, sizeof(uint64_t));
count_5845386402c0<<<std::ceil((float)orders_size/32.), 32>>>(d_COUNT584538630340, orders_size);
uint64_t COUNT584538630340;
hipMemcpy(&COUNT584538630340, d_COUNT584538630340, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT584538630340);
// Insert hash table control;
uint64_t* d_BUF_IDX_584538630340;
hipMalloc(&d_BUF_IDX_584538630340, sizeof(uint64_t));
hipMemset(d_BUF_IDX_584538630340, 0, sizeof(uint64_t));
uint64_t* d_BUF_584538630340;
hipMalloc(&d_BUF_584538630340, sizeof(uint64_t) * COUNT584538630340 * 1);
auto d_HT_584538630340 = cuco::experimental::static_multimap{ (int)COUNT584538630340*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5845386402c0<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_584538630340, d_BUF_IDX_584538630340, d_HT_584538630340.ref(cuco::insert), d_orders__o_orderkey, orders_size);
hipFree(d_BUF_IDX_584538630340);
//Materialize count
uint64_t* d_COUNT584538630450;
hipMalloc(&d_COUNT584538630450, sizeof(uint64_t));
hipMemset(d_COUNT584538630450, 0, sizeof(uint64_t));
count_584538533fd0<<<std::ceil((float)part_size/32.), 32>>>(d_COUNT584538630450, part_size);
uint64_t COUNT584538630450;
hipMemcpy(&COUNT584538630450, d_COUNT584538630450, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT584538630450);
// Insert hash table control;
uint64_t* d_BUF_IDX_584538630450;
hipMalloc(&d_BUF_IDX_584538630450, sizeof(uint64_t));
hipMemset(d_BUF_IDX_584538630450, 0, sizeof(uint64_t));
uint64_t* d_BUF_584538630450;
hipMalloc(&d_BUF_584538630450, sizeof(uint64_t) * COUNT584538630450 * 1);
auto d_HT_584538630450 = cuco::experimental::static_multimap{ (int)COUNT584538630450*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_584538533fd0<<<std::ceil((float)part_size/32.), 32>>>(d_BUF_584538630450, d_BUF_IDX_584538630450, d_HT_584538630450.ref(cuco::insert), d_part__p_partkey, part_size);
hipFree(d_BUF_IDX_584538630450);
//Materialize count
uint64_t* d_COUNT5845386305c0;
hipMalloc(&d_COUNT5845386305c0, sizeof(uint64_t));
hipMemset(d_COUNT5845386305c0, 0, sizeof(uint64_t));
count_584538639f70<<<std::ceil((float)partsupp_size/32.), 32>>>(d_BUF_58453862ccc0, d_COUNT5845386305c0, d_HT_58453862ccc0.ref(cuco::for_each), d_partsupp__ps_suppkey, partsupp_size);
uint64_t COUNT5845386305c0;
hipMemcpy(&COUNT5845386305c0, d_COUNT5845386305c0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5845386305c0);
// Insert hash table control;
uint64_t* d_BUF_IDX_5845386305c0;
hipMalloc(&d_BUF_IDX_5845386305c0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5845386305c0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5845386305c0;
hipMalloc(&d_BUF_5845386305c0, sizeof(uint64_t) * COUNT5845386305c0 * 3);
auto d_HT_5845386305c0 = cuco::experimental::static_multimap{ (int)COUNT5845386305c0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_584538639f70<<<std::ceil((float)partsupp_size/32.), 32>>>(d_BUF_58453862ccc0, d_BUF_5845386305c0, d_BUF_IDX_5845386305c0, d_HT_58453862ccc0.ref(cuco::for_each), d_HT_5845386305c0.ref(cuco::insert), d_partsupp__ps_partkey, d_partsupp__ps_suppkey, partsupp_size, d_supplier__s_suppkey);
hipFree(d_BUF_IDX_5845386305c0);
//Create aggregation hash table
auto d_HT_5845385e9110 = cuco::static_map{ (int)48009721*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_584538637b20<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_584538630340, d_BUF_584538630450, d_BUF_5845386305c0, d_HT_5845385e9110.ref(cuco::insert), d_HT_584538630340.ref(cuco::for_each), d_HT_584538630450.ref(cuco::for_each), d_HT_5845386305c0.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_suppkey, lineitem_size, d_orders__o_orderdate);
size_t COUNT5845385e9110 = d_HT_5845385e9110.size();
thrust::device_vector<int64_t> keys_5845385e9110(COUNT5845385e9110), vals_5845385e9110(COUNT5845385e9110);
d_HT_5845385e9110.retrieve_all(keys_5845385e9110.begin(), vals_5845385e9110.begin());
thrust::host_vector<int64_t> h_keys_5845385e9110(COUNT5845385e9110);
thrust::copy(keys_5845385e9110.begin(), keys_5845385e9110.end(), h_keys_5845385e9110.begin());
thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_5845385e9110(COUNT5845385e9110);
for (int i=0; i < COUNT5845385e9110; i++)
{actual_dict_5845385e9110[i] = cuco::make_pair(h_keys_5845385e9110[i], i);}
d_HT_5845385e9110.clear();
d_HT_5845385e9110.insert(actual_dict_5845385e9110.begin(), actual_dict_5845385e9110.end());
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT5845385e9110);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT5845385e9110);
DBI64Type* d_KEY_5845385e9110map0__tmp_attr0;
hipMalloc(&d_KEY_5845385e9110map0__tmp_attr0, sizeof(DBI64Type) * COUNT5845385e9110);
hipMemset(d_KEY_5845385e9110map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT5845385e9110);
main_584538637b20<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_584538630340, d_BUF_584538630450, d_BUF_5845386305c0, d_HT_5845385e9110.ref(cuco::find), d_HT_584538630340.ref(cuco::for_each), d_HT_584538630450.ref(cuco::for_each), d_HT_5845386305c0.ref(cuco::for_each), d_KEY_5845385e9110map0__tmp_attr0, d_aggr0__tmp_attr2, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_quantity, d_lineitem__l_suppkey, lineitem_size, d_orders__o_orderdate, d_partsupp__ps_supplycost);
//Materialize count
uint64_t* d_COUNT5845385fc300;
hipMalloc(&d_COUNT5845385fc300, sizeof(uint64_t));
hipMemset(d_COUNT5845385fc300, 0, sizeof(uint64_t));
count_58453864e8c0<<<std::ceil((float)COUNT5845385e9110/32.), 32>>>(COUNT5845385e9110, d_COUNT5845385fc300);
uint64_t COUNT5845385fc300;
hipMemcpy(&COUNT5845385fc300, d_COUNT5845385fc300, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5845385fc300);
//Materialize buffers
uint64_t* d_MAT_IDX5845385fc300;
hipMalloc(&d_MAT_IDX5845385fc300, sizeof(uint64_t));
hipMemset(d_MAT_IDX5845385fc300, 0, sizeof(uint64_t));
auto MAT5845385fc300map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT5845385fc300);
DBI64Type* d_MAT5845385fc300map0__tmp_attr0;
hipMalloc(&d_MAT5845385fc300map0__tmp_attr0, sizeof(DBI64Type) * COUNT5845385fc300);
auto MAT5845385fc300aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5845385fc300);
DBDecimalType* d_MAT5845385fc300aggr0__tmp_attr2;
hipMalloc(&d_MAT5845385fc300aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT5845385fc300);
main_58453864e8c0<<<std::ceil((float)COUNT5845385e9110/32.), 32>>>(COUNT5845385e9110, d_MAT5845385fc300aggr0__tmp_attr2, d_MAT5845385fc300map0__tmp_attr0, d_MAT_IDX5845385fc300, d_aggr0__tmp_attr2, d_KEY_5845385e9110map0__tmp_attr0);
hipFree(d_MAT_IDX5845385fc300);
hipMemcpy(MAT5845385fc300map0__tmp_attr0, d_MAT5845385fc300map0__tmp_attr0, sizeof(DBI64Type) * COUNT5845385fc300, hipMemcpyDeviceToHost);
hipMemcpy(MAT5845385fc300aggr0__tmp_attr2, d_MAT5845385fc300aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT5845385fc300, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5845385fc300; i++) { std::cout << MAT5845385fc300map0__tmp_attr0[i] << "\t";
std::cout << MAT5845385fc300aggr0__tmp_attr2[i] << "\t";
std::cout << std::endl; }
}