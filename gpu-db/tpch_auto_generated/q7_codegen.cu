#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5a3e586e76c0(uint64_t* COUNT5a3e586db7a0, DBStringType* n1___n_name, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n1___n_name = n1___n_name[tid];
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
//Materialize count
atomicAdd((int*)COUNT5a3e586db7a0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5a3e586e76c0(uint64_t* BUF_5a3e586db7a0, uint64_t* BUF_IDX_5a3e586db7a0, HASHTABLE_INSERT HT_5a3e586db7a0, DBStringType* n1___n_name, DBI32Type* n1___n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n1___n_name = n1___n_name[tid];
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
uint64_t KEY_5a3e586db7a0 = 0;
auto reg_n1___n_nationkey = n1___n_nationkey[tid];
KEY_5a3e586db7a0 <<= 32;
KEY_5a3e586db7a0 |= reg_n1___n_nationkey;
// Insert hash table kernel;
auto buf_idx_5a3e586db7a0 = atomicAdd((int*)BUF_IDX_5a3e586db7a0, 1);
HT_5a3e586db7a0.insert(cuco::pair{KEY_5a3e586db7a0, buf_idx_5a3e586db7a0});
BUF_5a3e586db7a0[buf_idx_5a3e586db7a0 * 1 + 0] = tid;
}
__global__ void count_5a3e586ea900(uint64_t* COUNT5a3e586dc300, DBStringType* n2___n_name, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n2___n_name = n2___n_name[tid];
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
//Materialize count
atomicAdd((int*)COUNT5a3e586dc300, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5a3e586ea900(uint64_t* BUF_5a3e586dc300, uint64_t* BUF_IDX_5a3e586dc300, HASHTABLE_INSERT HT_5a3e586dc300, DBStringType* n2___n_name, DBI32Type* n2___n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n2___n_name = n2___n_name[tid];
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
uint64_t KEY_5a3e586dc300 = 0;
auto reg_n2___n_nationkey = n2___n_nationkey[tid];
KEY_5a3e586dc300 <<= 32;
KEY_5a3e586dc300 |= reg_n2___n_nationkey;
// Insert hash table kernel;
auto buf_idx_5a3e586dc300 = atomicAdd((int*)BUF_IDX_5a3e586dc300, 1);
HT_5a3e586dc300.insert(cuco::pair{KEY_5a3e586dc300, buf_idx_5a3e586dc300});
BUF_5a3e586dc300[buf_idx_5a3e586dc300 * 1 + 0] = tid;
}
template<typename HASHTABLE_FIND>
__global__ void count_5a3e586eb880(uint64_t* BUF_5a3e586dc300, uint64_t* COUNT5a3e586de860, HASHTABLE_FIND HT_5a3e586dc300, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_5a3e586dc300 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];
KEY_5a3e586dc300 <<= 32;
KEY_5a3e586dc300 |= reg_customer__c_nationkey;
//Probe Hash table
auto SLOT_5a3e586dc300 = HT_5a3e586dc300.find(KEY_5a3e586dc300);
if (SLOT_5a3e586dc300 == HT_5a3e586dc300.end()) return;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5a3e586de860, 1);
}
template<typename HASHTABLE_FIND, typename HASHTABLE_INSERT>
__global__ void main_5a3e586eb880(uint64_t* BUF_5a3e586dc300, uint64_t* BUF_5a3e586de860, uint64_t* BUF_IDX_5a3e586de860, HASHTABLE_FIND HT_5a3e586dc300, HASHTABLE_INSERT HT_5a3e586de860, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_5a3e586dc300 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];
KEY_5a3e586dc300 <<= 32;
KEY_5a3e586dc300 |= reg_customer__c_nationkey;
//Probe Hash table
auto SLOT_5a3e586dc300 = HT_5a3e586dc300.find(KEY_5a3e586dc300);
if (SLOT_5a3e586dc300 == HT_5a3e586dc300.end()) return;
if (!(true)) return;
uint64_t KEY_5a3e586de860 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];
KEY_5a3e586de860 <<= 32;
KEY_5a3e586de860 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_5a3e586de860 = atomicAdd((int*)BUF_IDX_5a3e586de860, 1);
HT_5a3e586de860.insert(cuco::pair{KEY_5a3e586de860, buf_idx_5a3e586de860});
BUF_5a3e586de860[buf_idx_5a3e586de860 * 2 + 0] = BUF_5a3e586dc300[SLOT_5a3e586dc300->second * 1 + 0];
BUF_5a3e586de860[buf_idx_5a3e586de860 * 2 + 1] = tid;
}
template<typename HASHTABLE_FIND>
__global__ void count_5a3e586e61f0(uint64_t* BUF_5a3e586de860, uint64_t* COUNT5a3e586de970, HASHTABLE_FIND HT_5a3e586de860, DBI32Type* orders__o_custkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_5a3e586de860 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];
KEY_5a3e586de860 <<= 32;
KEY_5a3e586de860 |= reg_orders__o_custkey;
//Probe Hash table
auto SLOT_5a3e586de860 = HT_5a3e586de860.find(KEY_5a3e586de860);
if (SLOT_5a3e586de860 == HT_5a3e586de860.end()) return;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5a3e586de970, 1);
}
template<typename HASHTABLE_FIND, typename HASHTABLE_INSERT>
__global__ void main_5a3e586e61f0(uint64_t* BUF_5a3e586de860, uint64_t* BUF_5a3e586de970, uint64_t* BUF_IDX_5a3e586de970, HASHTABLE_FIND HT_5a3e586de860, HASHTABLE_INSERT HT_5a3e586de970, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_5a3e586de860 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];
KEY_5a3e586de860 <<= 32;
KEY_5a3e586de860 |= reg_orders__o_custkey;
//Probe Hash table
auto SLOT_5a3e586de860 = HT_5a3e586de860.find(KEY_5a3e586de860);
if (SLOT_5a3e586de860 == HT_5a3e586de860.end()) return;
if (!(true)) return;
uint64_t KEY_5a3e586de970 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];
KEY_5a3e586de970 <<= 32;
KEY_5a3e586de970 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_5a3e586de970 = atomicAdd((int*)BUF_IDX_5a3e586de970, 1);
HT_5a3e586de970.insert(cuco::pair{KEY_5a3e586de970, buf_idx_5a3e586de970});
BUF_5a3e586de970[buf_idx_5a3e586de970 * 2 + 0] = BUF_5a3e586de860[SLOT_5a3e586de860->second * 2 + 0];
BUF_5a3e586de970[buf_idx_5a3e586de970 * 2 + 1] = tid;
}
template<typename HASHTABLE_FIND>
__global__ void count_5a3e585dcd70(uint64_t* BUF_5a3e586db7a0, uint64_t* COUNT5a3e586dea80, HASHTABLE_FIND HT_5a3e586db7a0, DBI32Type* supplier__s_nationkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_5a3e586db7a0 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];
KEY_5a3e586db7a0 <<= 32;
KEY_5a3e586db7a0 |= reg_supplier__s_nationkey;
//Probe Hash table
auto SLOT_5a3e586db7a0 = HT_5a3e586db7a0.find(KEY_5a3e586db7a0);
if (SLOT_5a3e586db7a0 == HT_5a3e586db7a0.end()) return;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5a3e586dea80, 1);
}
template<typename HASHTABLE_FIND, typename HASHTABLE_INSERT>
__global__ void main_5a3e585dcd70(uint64_t* BUF_5a3e586db7a0, uint64_t* BUF_5a3e586dea80, uint64_t* BUF_IDX_5a3e586dea80, HASHTABLE_FIND HT_5a3e586db7a0, HASHTABLE_INSERT HT_5a3e586dea80, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_5a3e586db7a0 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];
KEY_5a3e586db7a0 <<= 32;
KEY_5a3e586db7a0 |= reg_supplier__s_nationkey;
//Probe Hash table
auto SLOT_5a3e586db7a0 = HT_5a3e586db7a0.find(KEY_5a3e586db7a0);
if (SLOT_5a3e586db7a0 == HT_5a3e586db7a0.end()) return;
if (!(true)) return;
uint64_t KEY_5a3e586dea80 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];
KEY_5a3e586dea80 <<= 32;
KEY_5a3e586dea80 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5a3e586dea80 = atomicAdd((int*)BUF_IDX_5a3e586dea80, 1);
HT_5a3e586dea80.insert(cuco::pair{KEY_5a3e586dea80, buf_idx_5a3e586dea80});
BUF_5a3e586dea80[buf_idx_5a3e586dea80 * 2 + 0] = BUF_5a3e586db7a0[SLOT_5a3e586db7a0->second * 1 + 0];
BUF_5a3e586dea80[buf_idx_5a3e586dea80 * 2 + 1] = tid;
}
template<typename HASHTABLE_FIND>
__global__ void count_5a3e586603c0(uint64_t* BUF_5a3e586de970, uint64_t* BUF_5a3e586dea80, uint64_t* COUNT5a3e586a5870, HASHTABLE_FIND HT_5a3e586de970, HASHTABLE_FIND HT_5a3e586dea80, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n1___n_name, DBStringType* n2___n_name) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9861, Predicate::lte))) return;
uint64_t KEY_5a3e586de970 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];
KEY_5a3e586de970 <<= 32;
KEY_5a3e586de970 |= reg_lineitem__l_orderkey;
//Probe Hash table
auto SLOT_5a3e586de970 = HT_5a3e586de970.find(KEY_5a3e586de970);
if (SLOT_5a3e586de970 == HT_5a3e586de970.end()) return;
if (!(true)) return;
uint64_t KEY_5a3e586dea80 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];
KEY_5a3e586dea80 <<= 32;
KEY_5a3e586dea80 |= reg_lineitem__l_suppkey;
//Probe Hash table
auto SLOT_5a3e586dea80 = HT_5a3e586dea80.find(KEY_5a3e586dea80);
if (SLOT_5a3e586dea80 == HT_5a3e586dea80.end()) return;
auto reg_n1___n_name = n1___n_name[BUF_5a3e586dea80[SLOT_5a3e586dea80->second * 2 + 0]];
auto reg_n2___n_name = n2___n_name[BUF_5a3e586de970[SLOT_5a3e586de970->second * 2 + 0]];
if (!((((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq))) || ((evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) && (true))) return;
//Materialize count
atomicAdd((int*)COUNT5a3e586a5870, 1);
}
template<typename HASHTABLE_FIND>
__global__ void main_5a3e586603c0(uint64_t* BUF_5a3e586de970, uint64_t* BUF_5a3e586dea80, HASHTABLE_FIND HT_5a3e586de970, HASHTABLE_FIND HT_5a3e586dea80, DBI64Type* MAT5a3e586a5870map0__tmp_attr0, DBDecimalType* MAT5a3e586a5870map0__tmp_attr1, DBStringType* MAT5a3e586a5870n1___n_name, DBStringType* MAT5a3e586a5870n2___n_name, uint64_t* MAT_IDX5a3e586a5870, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n1___n_name, DBStringType* n2___n_name) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9861, Predicate::lte))) return;
uint64_t KEY_5a3e586de970 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];
KEY_5a3e586de970 <<= 32;
KEY_5a3e586de970 |= reg_lineitem__l_orderkey;
//Probe Hash table
auto SLOT_5a3e586de970 = HT_5a3e586de970.find(KEY_5a3e586de970);
if (SLOT_5a3e586de970 == HT_5a3e586de970.end()) return;
if (!(true)) return;
uint64_t KEY_5a3e586dea80 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];
KEY_5a3e586dea80 <<= 32;
KEY_5a3e586dea80 |= reg_lineitem__l_suppkey;
//Probe Hash table
auto SLOT_5a3e586dea80 = HT_5a3e586dea80.find(KEY_5a3e586dea80);
if (SLOT_5a3e586dea80 == HT_5a3e586dea80.end()) return;
auto reg_n1___n_name = n1___n_name[BUF_5a3e586dea80[SLOT_5a3e586dea80->second * 2 + 0]];
auto reg_n2___n_name = n2___n_name[BUF_5a3e586de970[SLOT_5a3e586de970->second * 2 + 0]];
if (!((((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq))) || ((evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) && (true))) return;
//Materialize buffers
auto mat_idx5a3e586a5870 = atomicAdd((int*)MAT_IDX5a3e586a5870, 1);
MAT5a3e586a5870n1___n_name[mat_idx5a3e586a5870] = reg_n1___n_name;
MAT5a3e586a5870n2___n_name[mat_idx5a3e586a5870] = reg_n2___n_name;
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_lineitem__l_shipdate);
MAT5a3e586a5870map0__tmp_attr0[mat_idx5a3e586a5870] = reg_map0__tmp_attr0;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
MAT5a3e586a5870map0__tmp_attr1[mat_idx5a3e586a5870] = reg_map0__tmp_attr1;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT5a3e586db7a0;
hipMalloc(&d_COUNT5a3e586db7a0, sizeof(uint64_t));
hipMemset(d_COUNT5a3e586db7a0, 0, sizeof(uint64_t));
count_5a3e586e76c0<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT5a3e586db7a0, d_nation__n_name, nation_size);
uint64_t COUNT5a3e586db7a0;
hipMemcpy(&COUNT5a3e586db7a0, d_COUNT5a3e586db7a0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5a3e586db7a0);
// Insert hash table control;
uint64_t* d_BUF_IDX_5a3e586db7a0;
hipMalloc(&d_BUF_IDX_5a3e586db7a0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5a3e586db7a0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5a3e586db7a0;
hipMalloc(&d_BUF_5a3e586db7a0, sizeof(uint64_t) * COUNT5a3e586db7a0 * 1);
auto d_HT_5a3e586db7a0 = cuco::static_map{ (int)COUNT5a3e586db7a0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5a3e586e76c0<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_5a3e586db7a0, d_BUF_IDX_5a3e586db7a0, d_HT_5a3e586db7a0.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
hipFree(d_BUF_IDX_5a3e586db7a0);
//Materialize count
uint64_t* d_COUNT5a3e586dc300;
hipMalloc(&d_COUNT5a3e586dc300, sizeof(uint64_t));
hipMemset(d_COUNT5a3e586dc300, 0, sizeof(uint64_t));
count_5a3e586ea900<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT5a3e586dc300, d_nation__n_name, nation_size);
uint64_t COUNT5a3e586dc300;
hipMemcpy(&COUNT5a3e586dc300, d_COUNT5a3e586dc300, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5a3e586dc300);
// Insert hash table control;
uint64_t* d_BUF_IDX_5a3e586dc300;
hipMalloc(&d_BUF_IDX_5a3e586dc300, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5a3e586dc300, 0, sizeof(uint64_t));
uint64_t* d_BUF_5a3e586dc300;
hipMalloc(&d_BUF_5a3e586dc300, sizeof(uint64_t) * COUNT5a3e586dc300 * 1);
auto d_HT_5a3e586dc300 = cuco::static_map{ (int)COUNT5a3e586dc300*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5a3e586ea900<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_5a3e586dc300, d_BUF_IDX_5a3e586dc300, d_HT_5a3e586dc300.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
hipFree(d_BUF_IDX_5a3e586dc300);
//Materialize count
uint64_t* d_COUNT5a3e586de860;
hipMalloc(&d_COUNT5a3e586de860, sizeof(uint64_t));
hipMemset(d_COUNT5a3e586de860, 0, sizeof(uint64_t));
count_5a3e586eb880<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5a3e586dc300, d_COUNT5a3e586de860, d_HT_5a3e586dc300.ref(cuco::find), d_customer__c_nationkey, customer_size);
uint64_t COUNT5a3e586de860;
hipMemcpy(&COUNT5a3e586de860, d_COUNT5a3e586de860, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5a3e586de860);
// Insert hash table control;
uint64_t* d_BUF_IDX_5a3e586de860;
hipMalloc(&d_BUF_IDX_5a3e586de860, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5a3e586de860, 0, sizeof(uint64_t));
uint64_t* d_BUF_5a3e586de860;
hipMalloc(&d_BUF_5a3e586de860, sizeof(uint64_t) * COUNT5a3e586de860 * 2);
auto d_HT_5a3e586de860 = cuco::static_map{ (int)COUNT5a3e586de860*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5a3e586eb880<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5a3e586dc300, d_BUF_5a3e586de860, d_BUF_IDX_5a3e586de860, d_HT_5a3e586dc300.ref(cuco::find), d_HT_5a3e586de860.ref(cuco::insert), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
hipFree(d_BUF_IDX_5a3e586de860);
//Materialize count
uint64_t* d_COUNT5a3e586de970;
hipMalloc(&d_COUNT5a3e586de970, sizeof(uint64_t));
hipMemset(d_COUNT5a3e586de970, 0, sizeof(uint64_t));
count_5a3e586e61f0<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5a3e586de860, d_COUNT5a3e586de970, d_HT_5a3e586de860.ref(cuco::find), d_orders__o_custkey, orders_size);
uint64_t COUNT5a3e586de970;
hipMemcpy(&COUNT5a3e586de970, d_COUNT5a3e586de970, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5a3e586de970);
// Insert hash table control;
uint64_t* d_BUF_IDX_5a3e586de970;
hipMalloc(&d_BUF_IDX_5a3e586de970, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5a3e586de970, 0, sizeof(uint64_t));
uint64_t* d_BUF_5a3e586de970;
hipMalloc(&d_BUF_5a3e586de970, sizeof(uint64_t) * COUNT5a3e586de970 * 2);
auto d_HT_5a3e586de970 = cuco::static_map{ (int)COUNT5a3e586de970*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5a3e586e61f0<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5a3e586de860, d_BUF_5a3e586de970, d_BUF_IDX_5a3e586de970, d_HT_5a3e586de860.ref(cuco::find), d_HT_5a3e586de970.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderkey, orders_size);
hipFree(d_BUF_IDX_5a3e586de970);
//Materialize count
uint64_t* d_COUNT5a3e586dea80;
hipMalloc(&d_COUNT5a3e586dea80, sizeof(uint64_t));
hipMemset(d_COUNT5a3e586dea80, 0, sizeof(uint64_t));
count_5a3e585dcd70<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_5a3e586db7a0, d_COUNT5a3e586dea80, d_HT_5a3e586db7a0.ref(cuco::find), d_supplier__s_nationkey, supplier_size);
uint64_t COUNT5a3e586dea80;
hipMemcpy(&COUNT5a3e586dea80, d_COUNT5a3e586dea80, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5a3e586dea80);
// Insert hash table control;
uint64_t* d_BUF_IDX_5a3e586dea80;
hipMalloc(&d_BUF_IDX_5a3e586dea80, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5a3e586dea80, 0, sizeof(uint64_t));
uint64_t* d_BUF_5a3e586dea80;
hipMalloc(&d_BUF_5a3e586dea80, sizeof(uint64_t) * COUNT5a3e586dea80 * 2);
auto d_HT_5a3e586dea80 = cuco::static_map{ (int)COUNT5a3e586dea80*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5a3e585dcd70<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_5a3e586db7a0, d_BUF_5a3e586dea80, d_BUF_IDX_5a3e586dea80, d_HT_5a3e586db7a0.ref(cuco::find), d_HT_5a3e586dea80.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
hipFree(d_BUF_IDX_5a3e586dea80);
//Materialize count
uint64_t* d_COUNT5a3e586a5870;
hipMalloc(&d_COUNT5a3e586a5870, sizeof(uint64_t));
hipMemset(d_COUNT5a3e586a5870, 0, sizeof(uint64_t));
count_5a3e586603c0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5a3e586de970, d_BUF_5a3e586dea80, d_COUNT5a3e586a5870, d_HT_5a3e586de970.ref(cuco::find), d_HT_5a3e586dea80.ref(cuco::find), d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_nation__n_name);
uint64_t COUNT5a3e586a5870;
hipMemcpy(&COUNT5a3e586a5870, d_COUNT5a3e586a5870, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5a3e586a5870);
//Materialize buffers
uint64_t* d_MAT_IDX5a3e586a5870;
hipMalloc(&d_MAT_IDX5a3e586a5870, sizeof(uint64_t));
hipMemset(d_MAT_IDX5a3e586a5870, 0, sizeof(uint64_t));
auto MAT5a3e586a5870n1___n_name = (DBStringType*)malloc(sizeof(DBStringType) * COUNT5a3e586a5870);
DBStringType* d_MAT5a3e586a5870n1___n_name;
hipMalloc(&d_MAT5a3e586a5870n1___n_name, sizeof(DBStringType) * COUNT5a3e586a5870);
auto MAT5a3e586a5870n2___n_name = (DBStringType*)malloc(sizeof(DBStringType) * COUNT5a3e586a5870);
DBStringType* d_MAT5a3e586a5870n2___n_name;
hipMalloc(&d_MAT5a3e586a5870n2___n_name, sizeof(DBStringType) * COUNT5a3e586a5870);
auto MAT5a3e586a5870map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT5a3e586a5870);
DBI64Type* d_MAT5a3e586a5870map0__tmp_attr0;
hipMalloc(&d_MAT5a3e586a5870map0__tmp_attr0, sizeof(DBI64Type) * COUNT5a3e586a5870);
auto MAT5a3e586a5870map0__tmp_attr1 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5a3e586a5870);
DBDecimalType* d_MAT5a3e586a5870map0__tmp_attr1;
hipMalloc(&d_MAT5a3e586a5870map0__tmp_attr1, sizeof(DBDecimalType) * COUNT5a3e586a5870);
main_5a3e586603c0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5a3e586de970, d_BUF_5a3e586dea80, d_HT_5a3e586de970.ref(cuco::find), d_HT_5a3e586dea80.ref(cuco::find), d_MAT5a3e586a5870map0__tmp_attr0, d_MAT5a3e586a5870map0__tmp_attr1, d_MAT5a3e586a5870n1___n_name, d_MAT5a3e586a5870n2___n_name, d_MAT_IDX5a3e586a5870, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_nation__n_name);
hipFree(d_MAT_IDX5a3e586a5870);
hipMemcpy(MAT5a3e586a5870n1___n_name, d_MAT5a3e586a5870n1___n_name, sizeof(DBStringType) * COUNT5a3e586a5870, hipMemcpyDeviceToHost);
hipMemcpy(MAT5a3e586a5870n2___n_name, d_MAT5a3e586a5870n2___n_name, sizeof(DBStringType) * COUNT5a3e586a5870, hipMemcpyDeviceToHost);
hipMemcpy(MAT5a3e586a5870map0__tmp_attr0, d_MAT5a3e586a5870map0__tmp_attr0, sizeof(DBI64Type) * COUNT5a3e586a5870, hipMemcpyDeviceToHost);
hipMemcpy(MAT5a3e586a5870map0__tmp_attr1, d_MAT5a3e586a5870map0__tmp_attr1, sizeof(DBDecimalType) * COUNT5a3e586a5870, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5a3e586a5870; i++) { 
//     std::cout << MAT5a3e586a5870n1___n_name[i] << "\t";
// std::cout << MAT5a3e586a5870n2___n_name[i] << "\t";
std::cout << MAT5a3e586a5870map0__tmp_attr0[i] << "\t";
std::cout << MAT5a3e586a5870map0__tmp_attr1[i] << "\t";
std::cout << std::endl; }
}