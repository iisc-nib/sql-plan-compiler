#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_59c1d8837510(uint64_t* COUNT59c1d88a8480, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8674, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::lt))) return;
//Materialize count
atomicAdd((int*)COUNT59c1d88a8480, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_59c1d8837510(uint64_t* BUF_59c1d88a8480, uint64_t* BUF_IDX_59c1d88a8480, HASHTABLE_INSERT HT_59c1d88a8480, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8674, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::lt))) return;
uint64_t KEY_59c1d88a8480 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_59c1d88a8480 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_59c1d88a8480 = atomicAdd((int*)BUF_IDX_59c1d88a8480, 1);
HT_59c1d88a8480.insert(cuco::pair{KEY_59c1d88a8480, buf_idx_59c1d88a8480});
BUF_59c1d88a8480[buf_idx_59c1d88a8480 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_59c1d88b4a80(uint64_t* BUF_59c1d88a8480, uint64_t* COUNT59c1d88a7fa0, HASHTABLE_PROBE HT_59c1d88a8480, DBI32Type* lineitem__l_orderkey, DBCharType* lineitem__l_returnflag, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];
if (!(evaluatePredicate(reg_lineitem__l_returnflag, 'R', Predicate::eq))) return;
uint64_t KEY_59c1d88a8480 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_59c1d88a8480 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_59c1d88a8480.for_each(KEY_59c1d88a8480, [&] __device__ (auto const SLOT_59c1d88a8480) {

auto const [slot_first59c1d88a8480, slot_second59c1d88a8480] = SLOT_59c1d88a8480;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT59c1d88a7fa0, 1);
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_59c1d88b4a80(uint64_t* BUF_59c1d88a7fa0, uint64_t* BUF_59c1d88a8480, uint64_t* BUF_IDX_59c1d88a7fa0, HASHTABLE_INSERT HT_59c1d88a7fa0, HASHTABLE_PROBE HT_59c1d88a8480, DBI32Type* lineitem__l_orderkey, DBCharType* lineitem__l_returnflag, size_t lineitem_size, DBI32Type* orders__o_custkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];
if (!(evaluatePredicate(reg_lineitem__l_returnflag, 'R', Predicate::eq))) return;
uint64_t KEY_59c1d88a8480 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_59c1d88a8480 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_59c1d88a8480.for_each(KEY_59c1d88a8480, [&] __device__ (auto const SLOT_59c1d88a8480) {
auto const [slot_first59c1d88a8480, slot_second59c1d88a8480] = SLOT_59c1d88a8480;
if (!(true)) return;
uint64_t KEY_59c1d88a7fa0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[BUF_59c1d88a8480[slot_second59c1d88a8480 * 1 + 0]];

KEY_59c1d88a7fa0 |= reg_orders__o_custkey;
// Insert hash table kernel;
auto buf_idx_59c1d88a7fa0 = atomicAdd((int*)BUF_IDX_59c1d88a7fa0, 1);
HT_59c1d88a7fa0.insert(cuco::pair{KEY_59c1d88a7fa0, buf_idx_59c1d88a7fa0});
BUF_59c1d88a7fa0[buf_idx_59c1d88a7fa0 * 2 + 0] = BUF_59c1d88a8480[slot_second59c1d88a8480 * 1 + 0];
BUF_59c1d88a7fa0[buf_idx_59c1d88a7fa0 * 2 + 1] = tid;
});
}
__global__ void count_59c1d88bdbd0(uint64_t* COUNT59c1d88aac40, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
//Materialize count
atomicAdd((int*)COUNT59c1d88aac40, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_59c1d88bdbd0(uint64_t* BUF_59c1d88aac40, uint64_t* BUF_IDX_59c1d88aac40, HASHTABLE_INSERT HT_59c1d88aac40, DBI32Type* nation__n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_59c1d88aac40 = 0;
auto reg_nation__n_nationkey = nation__n_nationkey[tid];

KEY_59c1d88aac40 |= reg_nation__n_nationkey;
// Insert hash table kernel;
auto buf_idx_59c1d88aac40 = atomicAdd((int*)BUF_IDX_59c1d88aac40, 1);
HT_59c1d88aac40.insert(cuco::pair{KEY_59c1d88aac40, buf_idx_59c1d88aac40});
BUF_59c1d88aac40[buf_idx_59c1d88aac40 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_59c1d87b42c0(uint64_t* BUF_59c1d88a7fa0, uint64_t* BUF_59c1d88aac40, HASHTABLE_INSERT HT_59c1d8865cb0, HASHTABLE_PROBE HT_59c1d88a7fa0, HASHTABLE_PROBE HT_59c1d88aac40, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_59c1d88a7fa0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_59c1d88a7fa0 |= reg_customer__c_custkey;
//Probe Hash table
HT_59c1d88a7fa0.for_each(KEY_59c1d88a7fa0, [&] __device__ (auto const SLOT_59c1d88a7fa0) {

auto const [slot_first59c1d88a7fa0, slot_second59c1d88a7fa0] = SLOT_59c1d88a7fa0;
if (!(true)) return;
uint64_t KEY_59c1d88aac40 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_59c1d88aac40 |= reg_customer__c_nationkey;
//Probe Hash table
HT_59c1d88aac40.for_each(KEY_59c1d88aac40, [&] __device__ (auto const SLOT_59c1d88aac40) {

auto const [slot_first59c1d88aac40, slot_second59c1d88aac40] = SLOT_59c1d88aac40;
if (!(true)) return;
uint64_t KEY_59c1d8865cb0 = 0;

KEY_59c1d8865cb0 |= reg_customer__c_custkey;
//Create aggregation hash table
HT_59c1d8865cb0.insert(cuco::pair{KEY_59c1d8865cb0, 1});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_59c1d87b42c0(uint64_t* BUF_59c1d88a7fa0, uint64_t* BUF_59c1d88aac40, HASHTABLE_FIND HT_59c1d8865cb0, HASHTABLE_PROBE HT_59c1d88a7fa0, HASHTABLE_PROBE HT_59c1d88aac40, DBI32Type* KEY_59c1d8865cb0customer__c_custkey, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr__c_acctbal, DBStringType* aggr__c_address, DBStringType* aggr__c_comment, DBStringType* aggr__c_name, DBStringType* aggr__c_phone, DBStringType* aggr__n_name, DBDecimalType* customer__c_acctbal, DBStringType* customer__c_address, DBStringType* customer__c_comment, DBI32Type* customer__c_custkey, DBStringType* customer__c_name, DBI32Type* customer__c_nationkey, DBStringType* customer__c_phone, size_t customer_size, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBStringType* nation__n_name) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_59c1d88a7fa0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_59c1d88a7fa0 |= reg_customer__c_custkey;
//Probe Hash table
HT_59c1d88a7fa0.for_each(KEY_59c1d88a7fa0, [&] __device__ (auto const SLOT_59c1d88a7fa0) {
auto const [slot_first59c1d88a7fa0, slot_second59c1d88a7fa0] = SLOT_59c1d88a7fa0;
if (!(true)) return;
uint64_t KEY_59c1d88aac40 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_59c1d88aac40 |= reg_customer__c_nationkey;
//Probe Hash table
HT_59c1d88aac40.for_each(KEY_59c1d88aac40, [&] __device__ (auto const SLOT_59c1d88aac40) {
auto const [slot_first59c1d88aac40, slot_second59c1d88aac40] = SLOT_59c1d88aac40;
if (!(true)) return;
uint64_t KEY_59c1d8865cb0 = 0;

KEY_59c1d8865cb0 |= reg_customer__c_custkey;
//Aggregate in hashtable
auto buf_idx_59c1d8865cb0 = HT_59c1d8865cb0.find(KEY_59c1d8865cb0)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[BUF_59c1d88a7fa0[slot_second59c1d88a7fa0 * 2 + 1]];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[BUF_59c1d88a7fa0[slot_second59c1d88a7fa0 * 2 + 1]];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_59c1d8865cb0], reg_map0__tmp_attr1);
auto reg_customer__c_comment = customer__c_comment[tid];
aggregate_any(&aggr__c_comment[buf_idx_59c1d8865cb0], reg_customer__c_comment);
auto reg_customer__c_name = customer__c_name[tid];
aggregate_any(&aggr__c_name[buf_idx_59c1d8865cb0], reg_customer__c_name);
auto reg_customer__c_acctbal = customer__c_acctbal[tid];
aggregate_any(&aggr__c_acctbal[buf_idx_59c1d8865cb0], reg_customer__c_acctbal);
auto reg_customer__c_phone = customer__c_phone[tid];
aggregate_any(&aggr__c_phone[buf_idx_59c1d8865cb0], reg_customer__c_phone);
auto reg_nation__n_name = nation__n_name[BUF_59c1d88aac40[slot_second59c1d88aac40 * 1 + 0]];
aggregate_any(&aggr__n_name[buf_idx_59c1d8865cb0], reg_nation__n_name);
auto reg_customer__c_address = customer__c_address[tid];
aggregate_any(&aggr__c_address[buf_idx_59c1d8865cb0], reg_customer__c_address);
KEY_59c1d8865cb0customer__c_custkey[buf_idx_59c1d8865cb0] = reg_customer__c_custkey;
});
});
}
__global__ void count_59c1d88c4640(size_t COUNT59c1d8865cb0, uint64_t* COUNT59c1d8878190) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT59c1d8865cb0) return;
//Materialize count
atomicAdd((int*)COUNT59c1d8878190, 1);
}
__global__ void main_59c1d88c4640(size_t COUNT59c1d8865cb0, DBDecimalType* MAT59c1d8878190aggr0__tmp_attr0, DBDecimalType* MAT59c1d8878190aggr__c_acctbal, DBStringType* MAT59c1d8878190aggr__c_address, DBStringType* MAT59c1d8878190aggr__c_comment, DBStringType* MAT59c1d8878190aggr__c_name, DBStringType* MAT59c1d8878190aggr__c_phone, DBStringType* MAT59c1d8878190aggr__n_name, DBI32Type* MAT59c1d8878190customer__c_custkey, uint64_t* MAT_IDX59c1d8878190, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr__c_acctbal, DBStringType* aggr__c_address, DBStringType* aggr__c_comment, DBStringType* aggr__c_name, DBStringType* aggr__c_phone, DBStringType* aggr__n_name, DBI32Type* customer__c_custkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT59c1d8865cb0) return;
//Materialize buffers
auto mat_idx59c1d8878190 = atomicAdd((int*)MAT_IDX59c1d8878190, 1);
auto reg_customer__c_custkey = customer__c_custkey[tid];
MAT59c1d8878190customer__c_custkey[mat_idx59c1d8878190] = reg_customer__c_custkey;
auto reg_aggr__c_name = aggr__c_name[tid];
MAT59c1d8878190aggr__c_name[mat_idx59c1d8878190] = reg_aggr__c_name;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT59c1d8878190aggr0__tmp_attr0[mat_idx59c1d8878190] = reg_aggr0__tmp_attr0;
auto reg_aggr__c_acctbal = aggr__c_acctbal[tid];
MAT59c1d8878190aggr__c_acctbal[mat_idx59c1d8878190] = reg_aggr__c_acctbal;
auto reg_aggr__n_name = aggr__n_name[tid];
MAT59c1d8878190aggr__n_name[mat_idx59c1d8878190] = reg_aggr__n_name;
auto reg_aggr__c_address = aggr__c_address[tid];
MAT59c1d8878190aggr__c_address[mat_idx59c1d8878190] = reg_aggr__c_address;
auto reg_aggr__c_phone = aggr__c_phone[tid];
MAT59c1d8878190aggr__c_phone[mat_idx59c1d8878190] = reg_aggr__c_phone;
auto reg_aggr__c_comment = aggr__c_comment[tid];
MAT59c1d8878190aggr__c_comment[mat_idx59c1d8878190] = reg_aggr__c_comment;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT59c1d88a8480;
hipMalloc(&d_COUNT59c1d88a8480, sizeof(uint64_t));
hipMemset(d_COUNT59c1d88a8480, 0, sizeof(uint64_t));
count_59c1d8837510<<<std::ceil((float)orders_size/32.), 32>>>(d_COUNT59c1d88a8480, d_orders__o_orderdate, orders_size);
uint64_t COUNT59c1d88a8480;
hipMemcpy(&COUNT59c1d88a8480, d_COUNT59c1d88a8480, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT59c1d88a8480);
// Insert hash table control;
uint64_t* d_BUF_IDX_59c1d88a8480;
hipMalloc(&d_BUF_IDX_59c1d88a8480, sizeof(uint64_t));
hipMemset(d_BUF_IDX_59c1d88a8480, 0, sizeof(uint64_t));
uint64_t* d_BUF_59c1d88a8480;
hipMalloc(&d_BUF_59c1d88a8480, sizeof(uint64_t) * COUNT59c1d88a8480 * 1);
auto d_HT_59c1d88a8480 = cuco::experimental::static_multimap{ (int)COUNT59c1d88a8480*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_59c1d8837510<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_59c1d88a8480, d_BUF_IDX_59c1d88a8480, d_HT_59c1d88a8480.ref(cuco::insert), d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
hipFree(d_BUF_IDX_59c1d88a8480);
//Materialize count
uint64_t* d_COUNT59c1d88a7fa0;
hipMalloc(&d_COUNT59c1d88a7fa0, sizeof(uint64_t));
hipMemset(d_COUNT59c1d88a7fa0, 0, sizeof(uint64_t));
count_59c1d88b4a80<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_59c1d88a8480, d_COUNT59c1d88a7fa0, d_HT_59c1d88a8480.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_returnflag, lineitem_size);
uint64_t COUNT59c1d88a7fa0;
hipMemcpy(&COUNT59c1d88a7fa0, d_COUNT59c1d88a7fa0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT59c1d88a7fa0);
// Insert hash table control;
uint64_t* d_BUF_IDX_59c1d88a7fa0;
hipMalloc(&d_BUF_IDX_59c1d88a7fa0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_59c1d88a7fa0, 0, sizeof(uint64_t));
uint64_t* d_BUF_59c1d88a7fa0;
hipMalloc(&d_BUF_59c1d88a7fa0, sizeof(uint64_t) * COUNT59c1d88a7fa0 * 2);
auto d_HT_59c1d88a7fa0 = cuco::experimental::static_multimap{ (int)COUNT59c1d88a7fa0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_59c1d88b4a80<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_59c1d88a7fa0, d_BUF_59c1d88a8480, d_BUF_IDX_59c1d88a7fa0, d_HT_59c1d88a7fa0.ref(cuco::insert), d_HT_59c1d88a8480.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_returnflag, lineitem_size, d_orders__o_custkey);
hipFree(d_BUF_IDX_59c1d88a7fa0);
//Materialize count
uint64_t* d_COUNT59c1d88aac40;
hipMalloc(&d_COUNT59c1d88aac40, sizeof(uint64_t));
hipMemset(d_COUNT59c1d88aac40, 0, sizeof(uint64_t));
count_59c1d88bdbd0<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT59c1d88aac40, nation_size);
uint64_t COUNT59c1d88aac40;
hipMemcpy(&COUNT59c1d88aac40, d_COUNT59c1d88aac40, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT59c1d88aac40);
// Insert hash table control;
uint64_t* d_BUF_IDX_59c1d88aac40;
hipMalloc(&d_BUF_IDX_59c1d88aac40, sizeof(uint64_t));
hipMemset(d_BUF_IDX_59c1d88aac40, 0, sizeof(uint64_t));
uint64_t* d_BUF_59c1d88aac40;
hipMalloc(&d_BUF_59c1d88aac40, sizeof(uint64_t) * COUNT59c1d88aac40 * 1);
auto d_HT_59c1d88aac40 = cuco::experimental::static_multimap{ (int)COUNT59c1d88aac40*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_59c1d88bdbd0<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_59c1d88aac40, d_BUF_IDX_59c1d88aac40, d_HT_59c1d88aac40.ref(cuco::insert), d_nation__n_nationkey, nation_size);
hipFree(d_BUF_IDX_59c1d88aac40);
//Create aggregation hash table
auto d_HT_59c1d8865cb0 = cuco::static_map{ (int)45145*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_59c1d87b42c0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_59c1d88a7fa0, d_BUF_59c1d88aac40, d_HT_59c1d8865cb0.ref(cuco::insert), d_HT_59c1d88a7fa0.ref(cuco::for_each), d_HT_59c1d88aac40.ref(cuco::for_each), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
size_t COUNT59c1d8865cb0 = d_HT_59c1d8865cb0.size();
thrust::device_vector<int64_t> keys_59c1d8865cb0(COUNT59c1d8865cb0), vals_59c1d8865cb0(COUNT59c1d8865cb0);
d_HT_59c1d8865cb0.retrieve_all(keys_59c1d8865cb0.begin(), vals_59c1d8865cb0.begin());
thrust::host_vector<int64_t> h_keys_59c1d8865cb0(COUNT59c1d8865cb0);
thrust::copy(keys_59c1d8865cb0.begin(), keys_59c1d8865cb0.end(), h_keys_59c1d8865cb0.begin());
thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_59c1d8865cb0(COUNT59c1d8865cb0);
for (int i=0; i < COUNT59c1d8865cb0; i++)
{actual_dict_59c1d8865cb0[i] = cuco::make_pair(h_keys_59c1d8865cb0[i], i);}
d_HT_59c1d8865cb0.clear();
d_HT_59c1d8865cb0.insert(actual_dict_59c1d8865cb0.begin(), actual_dict_59c1d8865cb0.end());
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT59c1d8865cb0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT59c1d8865cb0);
DBStringType* d_aggr__c_comment;
hipMalloc(&d_aggr__c_comment, sizeof(DBStringType) * COUNT59c1d8865cb0);
hipMemset(d_aggr__c_comment, 0, sizeof(DBStringType) * COUNT59c1d8865cb0);
DBStringType* d_aggr__c_name;
hipMalloc(&d_aggr__c_name, sizeof(DBStringType) * COUNT59c1d8865cb0);
hipMemset(d_aggr__c_name, 0, sizeof(DBStringType) * COUNT59c1d8865cb0);
DBDecimalType* d_aggr__c_acctbal;
hipMalloc(&d_aggr__c_acctbal, sizeof(DBDecimalType) * COUNT59c1d8865cb0);
hipMemset(d_aggr__c_acctbal, 0, sizeof(DBDecimalType) * COUNT59c1d8865cb0);
DBStringType* d_aggr__c_phone;
hipMalloc(&d_aggr__c_phone, sizeof(DBStringType) * COUNT59c1d8865cb0);
hipMemset(d_aggr__c_phone, 0, sizeof(DBStringType) * COUNT59c1d8865cb0);
DBStringType* d_aggr__n_name;
hipMalloc(&d_aggr__n_name, sizeof(DBStringType) * COUNT59c1d8865cb0);
hipMemset(d_aggr__n_name, 0, sizeof(DBStringType) * COUNT59c1d8865cb0);
DBStringType* d_aggr__c_address;
hipMalloc(&d_aggr__c_address, sizeof(DBStringType) * COUNT59c1d8865cb0);
hipMemset(d_aggr__c_address, 0, sizeof(DBStringType) * COUNT59c1d8865cb0);
DBI32Type* d_KEY_59c1d8865cb0customer__c_custkey;
hipMalloc(&d_KEY_59c1d8865cb0customer__c_custkey, sizeof(DBI32Type) * COUNT59c1d8865cb0);
hipMemset(d_KEY_59c1d8865cb0customer__c_custkey, 0, sizeof(DBI32Type) * COUNT59c1d8865cb0);
main_59c1d87b42c0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_59c1d88a7fa0, d_BUF_59c1d88aac40, d_HT_59c1d8865cb0.ref(cuco::find), d_HT_59c1d88a7fa0.ref(cuco::for_each), d_HT_59c1d88aac40.ref(cuco::for_each), d_KEY_59c1d8865cb0customer__c_custkey, d_aggr0__tmp_attr0, d_aggr__c_acctbal, d_aggr__c_address, d_aggr__c_comment, d_aggr__c_name, d_aggr__c_phone, d_aggr__n_name, d_customer__c_acctbal, d_customer__c_address, d_customer__c_comment, d_customer__c_custkey, d_customer__c_name, d_customer__c_nationkey, d_customer__c_phone, customer_size, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_nation__n_name);
//Materialize count
uint64_t* d_COUNT59c1d8878190;
hipMalloc(&d_COUNT59c1d8878190, sizeof(uint64_t));
hipMemset(d_COUNT59c1d8878190, 0, sizeof(uint64_t));
count_59c1d88c4640<<<std::ceil((float)COUNT59c1d8865cb0/32.), 32>>>(COUNT59c1d8865cb0, d_COUNT59c1d8878190);
uint64_t COUNT59c1d8878190;
hipMemcpy(&COUNT59c1d8878190, d_COUNT59c1d8878190, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT59c1d8878190);
//Materialize buffers
uint64_t* d_MAT_IDX59c1d8878190;
hipMalloc(&d_MAT_IDX59c1d8878190, sizeof(uint64_t));
hipMemset(d_MAT_IDX59c1d8878190, 0, sizeof(uint64_t));
auto MAT59c1d8878190customer__c_custkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT59c1d8878190);
DBI32Type* d_MAT59c1d8878190customer__c_custkey;
hipMalloc(&d_MAT59c1d8878190customer__c_custkey, sizeof(DBI32Type) * COUNT59c1d8878190);
auto MAT59c1d8878190aggr__c_name = (DBStringType*)malloc(sizeof(DBStringType) * COUNT59c1d8878190);
DBStringType* d_MAT59c1d8878190aggr__c_name;
hipMalloc(&d_MAT59c1d8878190aggr__c_name, sizeof(DBStringType) * COUNT59c1d8878190);
auto MAT59c1d8878190aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT59c1d8878190);
DBDecimalType* d_MAT59c1d8878190aggr0__tmp_attr0;
hipMalloc(&d_MAT59c1d8878190aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT59c1d8878190);
auto MAT59c1d8878190aggr__c_acctbal = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT59c1d8878190);
DBDecimalType* d_MAT59c1d8878190aggr__c_acctbal;
hipMalloc(&d_MAT59c1d8878190aggr__c_acctbal, sizeof(DBDecimalType) * COUNT59c1d8878190);
auto MAT59c1d8878190aggr__n_name = (DBStringType*)malloc(sizeof(DBStringType) * COUNT59c1d8878190);
DBStringType* d_MAT59c1d8878190aggr__n_name;
hipMalloc(&d_MAT59c1d8878190aggr__n_name, sizeof(DBStringType) * COUNT59c1d8878190);
auto MAT59c1d8878190aggr__c_address = (DBStringType*)malloc(sizeof(DBStringType) * COUNT59c1d8878190);
DBStringType* d_MAT59c1d8878190aggr__c_address;
hipMalloc(&d_MAT59c1d8878190aggr__c_address, sizeof(DBStringType) * COUNT59c1d8878190);
auto MAT59c1d8878190aggr__c_phone = (DBStringType*)malloc(sizeof(DBStringType) * COUNT59c1d8878190);
DBStringType* d_MAT59c1d8878190aggr__c_phone;
hipMalloc(&d_MAT59c1d8878190aggr__c_phone, sizeof(DBStringType) * COUNT59c1d8878190);
auto MAT59c1d8878190aggr__c_comment = (DBStringType*)malloc(sizeof(DBStringType) * COUNT59c1d8878190);
DBStringType* d_MAT59c1d8878190aggr__c_comment;
hipMalloc(&d_MAT59c1d8878190aggr__c_comment, sizeof(DBStringType) * COUNT59c1d8878190);
main_59c1d88c4640<<<std::ceil((float)COUNT59c1d8865cb0/32.), 32>>>(COUNT59c1d8865cb0, d_MAT59c1d8878190aggr0__tmp_attr0, d_MAT59c1d8878190aggr__c_acctbal, d_MAT59c1d8878190aggr__c_address, d_MAT59c1d8878190aggr__c_comment, d_MAT59c1d8878190aggr__c_name, d_MAT59c1d8878190aggr__c_phone, d_MAT59c1d8878190aggr__n_name, d_MAT59c1d8878190customer__c_custkey, d_MAT_IDX59c1d8878190, d_aggr0__tmp_attr0, d_aggr__c_acctbal, d_aggr__c_address, d_aggr__c_comment, d_aggr__c_name, d_aggr__c_phone, d_aggr__n_name, d_KEY_59c1d8865cb0customer__c_custkey);
hipFree(d_MAT_IDX59c1d8878190);
hipMemcpy(MAT59c1d8878190customer__c_custkey, d_MAT59c1d8878190customer__c_custkey, sizeof(DBI32Type) * COUNT59c1d8878190, hipMemcpyDeviceToHost);
hipMemcpy(MAT59c1d8878190aggr__c_name, d_MAT59c1d8878190aggr__c_name, sizeof(DBStringType) * COUNT59c1d8878190, hipMemcpyDeviceToHost);
hipMemcpy(MAT59c1d8878190aggr0__tmp_attr0, d_MAT59c1d8878190aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT59c1d8878190, hipMemcpyDeviceToHost);
hipMemcpy(MAT59c1d8878190aggr__c_acctbal, d_MAT59c1d8878190aggr__c_acctbal, sizeof(DBDecimalType) * COUNT59c1d8878190, hipMemcpyDeviceToHost);
hipMemcpy(MAT59c1d8878190aggr__n_name, d_MAT59c1d8878190aggr__n_name, sizeof(DBStringType) * COUNT59c1d8878190, hipMemcpyDeviceToHost);
hipMemcpy(MAT59c1d8878190aggr__c_address, d_MAT59c1d8878190aggr__c_address, sizeof(DBStringType) * COUNT59c1d8878190, hipMemcpyDeviceToHost);
hipMemcpy(MAT59c1d8878190aggr__c_phone, d_MAT59c1d8878190aggr__c_phone, sizeof(DBStringType) * COUNT59c1d8878190, hipMemcpyDeviceToHost);
hipMemcpy(MAT59c1d8878190aggr__c_comment, d_MAT59c1d8878190aggr__c_comment, sizeof(DBStringType) * COUNT59c1d8878190, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT59c1d8878190; i++) { std::cout << MAT59c1d8878190customer__c_custkey[i] << "\t";
// std::cout << MAT59c1d8878190aggr__c_name[i] << "\t";
std::cout << MAT59c1d8878190aggr0__tmp_attr0[i] << "\t";
std::cout << MAT59c1d8878190aggr__c_acctbal[i] << "\t";
// std::cout << MAT59c1d8878190aggr__n_name[i] << "\t";
// std::cout << MAT59c1d8878190aggr__c_address[i] << "\t";
// std::cout << MAT59c1d8878190aggr__c_phone[i] << "\t";
// std::cout << MAT59c1d8878190aggr__c_comment[i] << "\t";
std::cout << std::endl; }
}