#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>
#include "cudautils.cuh"
#include "arrowutils.h"
#include <iomanip>
std::ostream& operator<<(std::ostream& o, const __int128& x) {
    if (x == std::numeric_limits<__int128>::min()) return o << "-170141183460469231731687303715884105728";
    if (x < 0) return o << "-" << -x;
    if (x < 10) return o << (char)(x + '0');
    return o << x / 10 << (char)(x % 10 + '0');
}

int main(int argc, const char**argv) {
    std::string dbDir = getDataDir(argv, argc);
    auto lineitem_table = getArrowTable(dbDir, "lineitem");
    PrintColumnTypes(lineitem_table);

    DBStringType* comments = readStringColumn(lineitem_table, "comments");
    for (auto i=0ull; i<lineitem_table->num_rows(); i++) {
        // std::cout << comments[i] << std::endl;
    }
    char* l_returnflag = readCharColumn(lineitem_table, "l_returnflag");
    DBDateType* l_shipdate = readDateColumn(lineitem_table, "l_shipdate");

    DBDecimalPrecisionType l_ep = readDecimalPrecisionColumn(lineitem_table, "l_extendedprice");
    DBDecimalType* l_epfloat = readDecimalColumn(lineitem_table, "l_extendedprice");

    std::cout << std::setprecision(12);
    for (auto i=0ull; i<lineitem_table->num_rows(); i++) {

        std::cout << (float)l_ep[i]/100.<< " " << l_epfloat[i] << std::endl;
    }
}
