#include "hip/hip_runtime.h"

#include "utils.h"

#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace cg = cooperative_groups;

__global__ void pipeline_1(int32_t *p_size, int32_t *p_partkey, char *p_type, int64_t *p_type_offsets, int *p_type_sizes, int32_t *ps_partkey, int64_t *B3_idx, size_t part_size)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= part_size)
        return;
    int32_t reg_p_size = p_size[tid];
    if (!(reg_p_size == 15))
        return;
    if (!(like_operator(p_type + p_type_offsets[tid], p_type_sizes[tid], "BRASS", 5)))
        return;
    int32_t reg_p_partkey = p_partkey[tid];
    int64_t key3 = 0;
    key3 |= (((int64_t)reg_p_partkey) << 0);
    atomicAdd((int *)B3_idx, 1);
}

template <typename TY_HT3_I, typename TY_HT3_F>
__global__ void pipeline_0(int32_t *p_size, int32_t *p_partkey, char *p_type, int64_t *p_type_offsets, int *p_type_sizes, int32_t *ps_partkey, TY_HT3_I HT3_I, TY_HT3_F HT3_F, int64_t *B3_part, int64_t *B3_idx, size_t part_size)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= part_size)
        return;
    int32_t reg_p_size = p_size[tid];
    if (!(reg_p_size == 15))
        return;
    if (!(like_operator(p_type + p_type_offsets[tid], p_type_sizes[tid], "BRASS", 5)))
        return;
    int32_t reg_p_partkey = p_partkey[tid];
    int64_t key3 = 0;
    key3 |= (((int64_t)reg_p_partkey) << 0);
    auto reg_B3_idx = atomicAdd((int *)B3_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT3_I.insert(thread, cuco::pair{key3, reg_B3_idx});
    B3_part[reg_B3_idx] = tid;
}

__global__ void pipeline_3(int8_t *r_name, int32_t *r_regionkey, int32_t *n_regionkey, int64_t *B0_idx, size_t region_size)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= region_size)
        return;
    int8_t reg_r_name = r_name[tid];
    if (!(reg_r_name == 3))
        return;
    int32_t reg_r_regionkey = r_regionkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_r_regionkey) << 0);
    atomicAdd((int *)B0_idx, 1);
}

template <typename TY_HT0_I, typename TY_HT0_F>
__global__ void pipeline_2(int32_t *r_regionkey, int32_t *n_regionkey, int8_t *r_name, TY_HT0_I HT0_I, TY_HT0_F HT0_F, int64_t *B0_idx, size_t region_size, int64_t *B0_region)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= region_size)
        return;
    int8_t reg_r_name = r_name[tid];
    if (!(reg_r_name == 3))
        return;
    int32_t reg_r_regionkey = r_regionkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_r_regionkey) << 0);
    auto reg_B0_idx = atomicAdd((int *)B0_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT0_I.insert(thread, cuco::pair{key0, reg_B0_idx});
    B0_region[reg_B0_idx] = tid;
}

template <typename TY_HT0_I, typename TY_HT0_F>
__global__ void pipeline_5(int32_t *n_nationkey, int32_t *s_nationkey, int32_t *n_regionkey, TY_HT0_I HT0_I, TY_HT0_F HT0_F, int64_t *B1_idx, int64_t *B0_region, size_t nation_size)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= nation_size)
        return;
    int32_t reg_n_regionkey = n_regionkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_n_regionkey) << 0);
    auto slot0 = HT0_F.find(key0);
    if (slot0 == HT0_F.end())
        return;
    int32_t reg_n_nationkey = n_nationkey[tid];
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_n_nationkey) << 0);
    atomicAdd((int *)B1_idx, 1);
}

template <typename TY_HT0_I, typename TY_HT0_F, typename TY_HT1_I, typename TY_HT1_F>
__global__ void pipeline_4(int32_t *n_nationkey, int32_t *s_nationkey, int32_t *n_regionkey, TY_HT0_I HT0_I, TY_HT0_F HT0_F, TY_HT1_I HT1_I, TY_HT1_F HT1_F, int64_t *B1_region, int64_t *B1_idx, int64_t *B1_nation, size_t nation_size, int64_t *B0_region)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= nation_size)
        return;
    int32_t reg_n_regionkey = n_regionkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_n_regionkey) << 0);
    auto slot0 = HT0_F.find(key0);
    if (slot0 == HT0_F.end())
        return;
    int32_t reg_n_nationkey = n_nationkey[tid];
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_n_nationkey) << 0);
    auto reg_B1_idx = atomicAdd((int *)B1_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT1_I.insert(thread, cuco::pair{key1, reg_B1_idx});
    B1_region[reg_B1_idx] = B0_region[slot0->second];
    B1_nation[reg_B1_idx] = tid;
}

template <typename TY_HT1_I, typename TY_HT1_F>
__global__ void pipeline_7(int32_t *ps_suppkey, int32_t *s_nationkey, int32_t *s_suppkey, TY_HT1_I HT1_I, TY_HT1_F HT1_F, size_t supplier_size, int64_t *B1_region, int64_t *B2_idx, int64_t *B1_nation)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= supplier_size)
        return;
    int32_t reg_s_nationkey = s_nationkey[tid];
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_s_nationkey) << 0);
    auto slot1 = HT1_F.find(key1);
    if (slot1 == HT1_F.end())
        return;
    int32_t reg_s_suppkey = s_suppkey[tid];
    int64_t key2 = 0;
    key2 |= (((int64_t)reg_s_suppkey) << 0);
    atomicAdd((int *)B2_idx, 1);
}

template <typename TY_HT1_I, typename TY_HT1_F, typename TY_HT2_I, typename TY_HT2_F>
__global__ void pipeline_6(int32_t *ps_suppkey, int32_t *s_nationkey, int32_t *s_suppkey, TY_HT1_I HT1_I, TY_HT1_F HT1_F, TY_HT2_I HT2_I, TY_HT2_F HT2_F, int64_t *B2_idx, int64_t *B2_supplier, int64_t *B2_nation, size_t supplier_size, int64_t *B1_nation, int64_t *B2_region, int64_t *B1_region)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= supplier_size)
        return;
    int32_t reg_s_nationkey = s_nationkey[tid];
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_s_nationkey) << 0);
    auto slot1 = HT1_F.find(key1);
    if (slot1 == HT1_F.end())
        return;
    int32_t reg_s_suppkey = s_suppkey[tid];
    int64_t key2 = 0;
    key2 |= (((int64_t)reg_s_suppkey) << 0);
    auto reg_B2_idx = atomicAdd((int *)B2_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT2_I.insert(thread, cuco::pair{key2, reg_B2_idx});
    B2_supplier[reg_B2_idx] = tid;
    B2_nation[reg_B2_idx] = B1_nation[slot1->second];
    B2_region[reg_B2_idx] = B1_region[slot1->second];
}

template <typename TY_HT2_I, typename TY_HT2_F, typename TY_HT3_I, typename TY_HT3_F, typename TY_HT4_I, typename TY_HT4_F>
__global__ void pipeline_8(int32_t *ps_suppkey, int32_t *ps_partkey, TY_HT2_I HT2_I, TY_HT2_F HT2_F, TY_HT3_I HT3_I, TY_HT3_F HT3_F, TY_HT4_I HT4_I, TY_HT4_F HT4_F, int64_t *B2_supplier, size_t partsupp_size, int64_t *B3_part, int64_t *B2_region, int64_t *B2_nation)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= partsupp_size)
        return;
    int32_t reg_ps_suppkey = ps_suppkey[tid];
    int64_t key2 = 0;
    key2 |= (((int64_t)reg_ps_suppkey) << 0);
    auto slot2 = HT2_F.find(key2);
    if (slot2 == HT2_F.end())
        return;
    int32_t reg_ps_partkey = ps_partkey[tid];
    int64_t key3 = 0;
    key3 |= (((int64_t)reg_ps_partkey) << 0);
    auto slot3 = HT3_F.find(key3);
    if (slot3 == HT3_F.end())
        return;
    int64_t key4 = 0;
    key4 |= (((int64_t)reg_ps_partkey) << 0);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT4_I.insert(thread, cuco::pair{key4, 1});
}

template <typename TY_HT2_I, typename TY_HT2_F, typename TY_HT3_I, typename TY_HT3_F, typename TY_HT4_I, typename TY_HT4_F>
__global__ void pipeline_9(double *ps_supplycost, int32_t *ps_suppkey, int32_t *ps_partkey, int8_t *p_mfgr, int32_t *p_partkey, int32_t *agg_p_partkey, int8_t *agg_p_mfgr, double *min_supplycost, TY_HT2_I HT2_I, TY_HT2_F HT2_F, TY_HT3_I HT3_I, TY_HT3_F HT3_F, TY_HT4_I HT4_I, TY_HT4_F HT4_F, int64_t *B2_supplier, size_t partsupp_size, int64_t *B3_part, int64_t *B2_region, int64_t *B2_nation)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= partsupp_size)
        return;
    int32_t reg_ps_suppkey = ps_suppkey[tid];
    int64_t key2 = 0;
    key2 |= (((int64_t)reg_ps_suppkey) << 0);
    auto slot2 = HT2_F.find(key2);
    if (slot2 == HT2_F.end())
        return;
    int32_t reg_ps_partkey = ps_partkey[tid];
    int64_t key3 = 0;
    key3 |= (((int64_t)reg_ps_partkey) << 0);
    auto slot3 = HT3_F.find(key3);
    if (slot3 == HT3_F.end())
        return;
    int64_t key4 = 0;
    key4 |= (((int64_t)reg_ps_partkey) << 0);
    auto slot4 = HT4_F.find(key4);
    double reg_ps_supplycost = ps_supplycost[tid];
    int8_t reg_p_mfgr = p_mfgr[B3_part[slot3->second]];
    int32_t reg_p_partkey = p_partkey[B3_part[slot3->second]];
    aggregate_min(&(min_supplycost[slot4->second]), reg_ps_supplycost);
    agg_p_mfgr[slot4->second] = reg_p_mfgr;
    agg_p_partkey[slot4->second] = reg_p_partkey;
}

__global__ void pipeline_11(int32_t *n_regionkey, int8_t *r_name, int32_t *r_regionkey, size_t region_size, int64_t *B7_idx)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= region_size)
        return;
    int8_t reg_r_name = r_name[tid];
    if (!(reg_r_name == 3))
        return;
    int32_t reg_r_regionkey = r_regionkey[tid];
    int64_t key7 = 0;
    key7 |= (((int64_t)reg_r_regionkey) << 0);
    atomicAdd((int *)B7_idx, 1);
}

template <typename TY_HT7_I, typename TY_HT7_F>
__global__ void pipeline_10(int32_t *n_regionkey, int8_t *r_name, int32_t *r_regionkey, TY_HT7_I HT7_I, TY_HT7_F HT7_F, int64_t *B7_region, size_t region_size, int64_t *B7_idx)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= region_size)
        return;
    int8_t reg_r_name = r_name[tid];
    if (!(reg_r_name == 3))
        return;
    int32_t reg_r_regionkey = r_regionkey[tid];
    int64_t key7 = 0;
    key7 |= (((int64_t)reg_r_regionkey) << 0);
    auto reg_B7_idx = atomicAdd((int *)B7_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT7_I.insert(thread, cuco::pair{key7, reg_B7_idx});
    B7_region[reg_B7_idx] = tid;
}

template <typename TY_HT7_I, typename TY_HT7_F>
__global__ void pipeline_13(int32_t *s_nationkey, int32_t *n_regionkey, int32_t *n_nationkey, TY_HT7_I HT7_I, TY_HT7_F HT7_F, int64_t *B8_idx, int64_t *B7_region, size_t nation_size)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= nation_size)
        return;
    int32_t reg_n_regionkey = n_regionkey[tid];
    int64_t key7 = 0;
    key7 |= (((int64_t)reg_n_regionkey) << 0);
    auto slot7 = HT7_F.find(key7);
    if (slot7 == HT7_F.end())
        return;
    int32_t reg_n_nationkey = n_nationkey[tid];
    int64_t key8 = 0;
    key8 |= (((int64_t)reg_n_nationkey) << 0);
    atomicAdd((int *)B8_idx, 1);
}

template <typename TY_HT8_I, typename TY_HT8_F, typename TY_HT7_I, typename TY_HT7_F>
__global__ void pipeline_12(int32_t *s_nationkey, int32_t *n_regionkey, int32_t *n_nationkey, TY_HT8_I HT8_I, TY_HT8_F HT8_F, TY_HT7_I HT7_I, TY_HT7_F HT7_F, int64_t *B7_region, int64_t *B8_region, int64_t *B8_nation, int64_t *B8_idx, size_t nation_size)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= nation_size)
        return;
    int32_t reg_n_regionkey = n_regionkey[tid];
    int64_t key7 = 0;
    key7 |= (((int64_t)reg_n_regionkey) << 0);
    auto slot7 = HT7_F.find(key7);
    if (slot7 == HT7_F.end())
        return;
    int32_t reg_n_nationkey = n_nationkey[tid];
    int64_t key8 = 0;
    key8 |= (((int64_t)reg_n_nationkey) << 0);
    auto reg_B8_idx = atomicAdd((int *)B8_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT8_I.insert(thread, cuco::pair{key8, reg_B8_idx});
    B8_region[reg_B8_idx] = B7_region[slot7->second];
    B8_nation[reg_B8_idx] = tid;
}

__global__ void pipeline_15(int32_t *s_suppkey, int32_t *ps_suppkey, size_t supplier_size, int64_t *B6_idx)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= supplier_size)
        return;
    int32_t reg_s_suppkey = s_suppkey[tid];
    int64_t key6 = 0;
    key6 |= (((int64_t)reg_s_suppkey) << 0);
    atomicAdd((int *)B6_idx, 1);
}

template <typename TY_HT6_I, typename TY_HT6_F>
__global__ void pipeline_14(int32_t *s_suppkey, int32_t *ps_suppkey, TY_HT6_I HT6_I, TY_HT6_F HT6_F, size_t supplier_size, int64_t *B6_supplier, int64_t *B6_idx)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= supplier_size)
        return;
    int32_t reg_s_suppkey = s_suppkey[tid];
    int64_t key6 = 0;
    key6 |= (((int64_t)reg_s_suppkey) << 0);
    auto reg_B6_idx = atomicAdd((int *)B6_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT6_I.insert(thread, cuco::pair{key6, reg_B6_idx});
    B6_supplier[reg_B6_idx] = tid;
}

__global__ void pipeline_17(double *ps_supplycost, int32_t *ps_partkey, int32_t *agg_p_partkey, double *min_supplycost, size_t agg_size, int64_t *B5_idx)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= agg_size)
        return;
    int32_t reg_agg_p_partkey = agg_p_partkey[tid];
    double reg_min_supplycost = min_supplycost[tid];
    int64_t key5 = 0;
    key5 |= (((int64_t)reg_agg_p_partkey) << 0);
    key5 |= (((int64_t)__double_as_longlong(reg_min_supplycost)) << 32);
    atomicAdd((int *)B5_idx, 1);
}

template <typename TY_HT5_I, typename TY_HT5_F>
__global__ void pipeline_16(double *ps_supplycost, int32_t *ps_partkey, int32_t *agg_p_partkey, double *min_supplycost, TY_HT5_I HT5_I, TY_HT5_F HT5_F, size_t agg_size, int64_t *B5_agg, int64_t *B5_idx)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= agg_size)
        return;
    int32_t reg_agg_p_partkey = agg_p_partkey[tid];
    float reg_min_supplycost = (float)min_supplycost[tid];
    int64_t key5 = 0;
    key5 |= (((int64_t)reg_agg_p_partkey) << 0);
    key5 |= (((int64_t)__double_as_longlong(reg_min_supplycost)) << 32);
    auto reg_B5_idx = atomicAdd((int *)B5_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT5_I.insert(thread, cuco::pair{key5, reg_B5_idx});
    B5_agg[reg_B5_idx] = tid;
}

template <typename TY_HT8_I, typename TY_HT8_F, typename TY_HT5_I, typename TY_HT5_F, typename TY_HT6_I, typename TY_HT6_F>
__global__ void pipeline_18(double *ps_supplycost, int32_t *s_nationkey, int32_t *ps_suppkey, int32_t *ps_partkey, TY_HT8_I HT8_I, TY_HT8_F HT8_F, TY_HT5_I HT5_I, TY_HT5_F HT5_F, TY_HT6_I HT6_I, TY_HT6_F HT6_F, int64_t *B5_agg, int64_t *B6_supplier, int64_t *B8_region, int64_t *B8_nation, size_t partsupp_size, int64_t *d_mat_idx9)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= partsupp_size)
        return;
    int32_t reg_ps_partkey = ps_partkey[tid];
    float reg_ps_supplycost = (float)ps_supplycost[tid];
    int64_t key5 = 0;
    key5 |= (((int64_t)reg_ps_partkey) << 0);
    key5 |= (((int64_t)__double_as_longlong(reg_ps_supplycost)) << 32);
    auto slot5 = HT5_F.find(key5);
    if (slot5 == HT5_F.end())
        return;
    int32_t reg_ps_suppkey = ps_suppkey[tid];
    int64_t key6 = 0;
    key6 |= (((int64_t)reg_ps_suppkey) << 0);
    auto slot6 = HT6_F.find(key6);
    if (slot6 == HT6_F.end())
        return;
    int32_t reg_s_nationkey = s_nationkey[B6_supplier[slot6->second]];
    int64_t key8 = 0;
    key8 |= (((int64_t)reg_s_nationkey) << 0);
    auto slot8 = HT8_F.find(key8);
    if (slot8 == HT8_F.end())
        return;
    atomicAdd((int *)d_mat_idx9, 1);
}

template <typename TY_HT8_I, typename TY_HT8_F, typename TY_HT5_I, typename TY_HT5_F, typename TY_HT6_I, typename TY_HT6_F>
__global__ void pipeline_19(double *s_acctbal, int32_t *ps_suppkey, int32_t *s_nationkey, double *ps_supplycost, int8_t *n_name, int8_t *agg_p_mfgr, int32_t *agg_p_partkey, int32_t *ps_partkey, double *mat_s_acctbal, int32_t *mat_p_partkey, int8_t *mat_p_mfgr, int8_t *mat_n_name, TY_HT8_I HT8_I, TY_HT8_F HT8_F, TY_HT5_I HT5_I, TY_HT5_F HT5_F, TY_HT6_I HT6_I, TY_HT6_F HT6_F, int64_t *B5_agg, int64_t *B6_supplier, int64_t *B8_region, int64_t *B8_nation, size_t partsupp_size, int64_t *d_mat_idx9)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= partsupp_size)
        return;
    int32_t reg_ps_partkey = ps_partkey[tid];
    float reg_ps_supplycost = (float)ps_supplycost[tid];
    int64_t key5 = 0;
    key5 |= (((int64_t)reg_ps_partkey) << 0);
    key5 |= (((int64_t)__double_as_longlong(reg_ps_supplycost)) << 32);
    auto slot5 = HT5_F.find(key5);
    if (slot5 == HT5_F.end())
        return;
    int32_t reg_ps_suppkey = ps_suppkey[tid];
    int64_t key6 = 0;
    key6 |= (((int64_t)reg_ps_suppkey) << 0);
    auto slot6 = HT6_F.find(key6);
    if (slot6 == HT6_F.end())
        return;
    int32_t reg_s_nationkey = s_nationkey[B6_supplier[slot6->second]];
    int64_t key8 = 0;
    key8 |= (((int64_t)reg_s_nationkey) << 0);
    auto slot8 = HT8_F.find(key8);
    if (slot8 == HT8_F.end())
        return;
    double reg_s_acctbal = s_acctbal[B6_supplier[slot6->second]];
    int8_t reg_n_name = n_name[B8_nation[slot8->second]];
    int32_t reg_agg_p_partkey = agg_p_partkey[B5_agg[slot5->second]];
    int8_t reg_agg_p_mfgr = agg_p_mfgr[B5_agg[slot5->second]];
    auto tmp_idx9 = atomicAdd((int *)d_mat_idx9, 1); 
    mat_s_acctbal[tmp_idx9] = reg_s_acctbal;
    mat_n_name[tmp_idx9] = reg_n_name;
    mat_p_partkey[tmp_idx9] = reg_agg_p_partkey;
    mat_p_mfgr[tmp_idx9] = reg_agg_p_mfgr;
}

void control(
    int8_t *n_name,
    int32_t *n_nationkey,
    int32_t *n_regionkey,
    int8_t *p_mfgr,
    int32_t *p_partkey,
    int32_t *p_size,
    StringColumn *p_type,
    int32_t *ps_partkey,
    int32_t *ps_suppkey,
    double *ps_supplycost,
    int8_t *r_name,
    int32_t *r_regionkey,
    int32_t *s_nationkey,
    double *s_acctbal,
    int32_t *s_suppkey,
    size_t nation_size,
    size_t part_size,
    size_t partsupp_size,
    size_t region_size,
    size_t supplier_size)
{
    int32_t *d_p_size;

    hipMalloc(&d_p_size, sizeof(int32_t) * part_size);

    hipMemcpy(d_p_size, p_size, sizeof(int32_t) * part_size, hipMemcpyHostToDevice);

    int32_t *d_p_partkey;

    hipMalloc(&d_p_partkey, sizeof(int32_t) * part_size);

    hipMemcpy(d_p_partkey, p_partkey, sizeof(int32_t) * part_size, hipMemcpyHostToDevice);

    char *d_p_type;

    int64_t *d_p_type_offsets;

    int *d_p_type_sizes;

    hipMalloc(&d_p_type, sizeof(char) * (p_type->offsets[part_size - 1] + p_type->sizes[part_size - 1]));

    hipMemcpy(d_p_type, p_type->data, sizeof(char) * (p_type->offsets[part_size - 1] + p_type->sizes[part_size - 1]), hipMemcpyHostToDevice);

    hipMalloc(&d_p_type_offsets, sizeof(int64_t) * part_size);

    hipMemcpy(d_p_type_offsets, p_type->offsets, sizeof(int64_t) * part_size, hipMemcpyHostToDevice);

    hipMalloc(&d_p_type_sizes, sizeof(int) * part_size);

    hipMemcpy(d_p_type_sizes, p_type->sizes, sizeof(int) * part_size, hipMemcpyHostToDevice);

    int32_t *d_ps_partkey;

    hipMalloc(&d_ps_partkey, sizeof(int32_t) * partsupp_size);

    hipMemcpy(d_ps_partkey, ps_partkey, sizeof(int32_t) * partsupp_size, hipMemcpyHostToDevice);

    int64_t *B3_part;
    int64_t *B3_idx;
    hipMalloc(&B3_idx, sizeof(int64_t));
    hipMemset(B3_idx, 0, sizeof(int64_t));
    pipeline_1<<<std::ceil((float)part_size / (float)32), 32>>>(d_p_size, d_p_partkey, d_p_type, d_p_type_offsets, d_p_type_sizes, d_ps_partkey, B3_idx, part_size);

    int64_t h_B3_idx;
    hipMemcpy(&h_B3_idx, B3_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B3_idx, 0, sizeof(int64_t));
    hipMalloc(&B3_part, sizeof(int64_t) * h_B3_idx);
    auto HT3 = cuco::static_map{h_B3_idx * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT3_F = HT3.ref(cuco::find);

    auto d_HT3_I = HT3.ref(cuco::insert);

    pipeline_0<<<std::ceil((float)part_size / (float)32), 32>>>(d_p_size, d_p_partkey, d_p_type, d_p_type_offsets, d_p_type_sizes, d_ps_partkey, d_HT3_I, d_HT3_F, B3_part, B3_idx, part_size);

    int32_t *d_r_regionkey;

    hipMalloc(&d_r_regionkey, sizeof(int32_t) * region_size);

    hipMemcpy(d_r_regionkey, r_regionkey, sizeof(int32_t) * region_size, hipMemcpyHostToDevice);

    int32_t *d_n_regionkey;

    hipMalloc(&d_n_regionkey, sizeof(int32_t) * nation_size);

    hipMemcpy(d_n_regionkey, n_regionkey, sizeof(int32_t) * nation_size, hipMemcpyHostToDevice);

    int8_t *d_r_name;

    hipMalloc(&d_r_name, sizeof(int8_t) * region_size);

    hipMemcpy(d_r_name, r_name, sizeof(int8_t) * region_size, hipMemcpyHostToDevice);

    int64_t *B0_region;
    int64_t *B0_idx;
    hipMalloc(&B0_idx, sizeof(int64_t));
    hipMemset(B0_idx, 0, sizeof(int64_t));
    pipeline_3<<<std::ceil((float)region_size / (float)32), 32>>>(d_r_name, d_r_regionkey, d_n_regionkey, B0_idx, region_size);

    int64_t h_B0_idx;
    hipMemcpy(&h_B0_idx, B0_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B0_idx, 0, sizeof(int64_t));
    hipMalloc(&B0_region, sizeof(int64_t) * h_B0_idx);
    auto HT0 = cuco::static_map{h_B0_idx * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT0_F = HT0.ref(cuco::find);

    auto d_HT0_I = HT0.ref(cuco::insert);

    pipeline_2<<<std::ceil((float)region_size / (float)32), 32>>>(d_r_regionkey, d_n_regionkey, d_r_name, d_HT0_I, d_HT0_F, B0_idx, region_size, B0_region);

    int32_t *d_n_nationkey;

    hipMalloc(&d_n_nationkey, sizeof(int32_t) * nation_size);

    hipMemcpy(d_n_nationkey, n_nationkey, sizeof(int32_t) * nation_size, hipMemcpyHostToDevice);

    int32_t *d_s_nationkey;

    hipMalloc(&d_s_nationkey, sizeof(int32_t) * supplier_size);

    hipMemcpy(d_s_nationkey, s_nationkey, sizeof(int32_t) * supplier_size, hipMemcpyHostToDevice);

    int64_t *B1_region;
    int64_t *B1_nation;
    int64_t *B1_idx;
    hipMalloc(&B1_idx, sizeof(int64_t));
    hipMemset(B1_idx, 0, sizeof(int64_t));
    pipeline_5<<<std::ceil((float)nation_size / (float)32), 32>>>(d_n_nationkey, d_s_nationkey, d_n_regionkey, d_HT0_I, d_HT0_F, B1_idx, B0_region, nation_size);

    int64_t h_B1_idx;
    hipMemcpy(&h_B1_idx, B1_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B1_idx, 0, sizeof(int64_t));
    hipMalloc(&B1_nation, sizeof(int64_t) * h_B1_idx);
    hipMalloc(&B1_region, sizeof(int64_t) * h_B1_idx);
    auto HT1 = cuco::static_map{h_B1_idx * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT1_F = HT1.ref(cuco::find);

    auto d_HT1_I = HT1.ref(cuco::insert);

    pipeline_4<<<std::ceil((float)nation_size / (float)32), 32>>>(d_n_nationkey, d_s_nationkey, d_n_regionkey, d_HT0_I, d_HT0_F, d_HT1_I, d_HT1_F, B1_region, B1_idx, B1_nation, nation_size, B0_region);

    int32_t *d_ps_suppkey;

    hipMalloc(&d_ps_suppkey, sizeof(int32_t) * partsupp_size);

    hipMemcpy(d_ps_suppkey, ps_suppkey, sizeof(int32_t) * partsupp_size, hipMemcpyHostToDevice);

    int32_t *d_s_suppkey;

    hipMalloc(&d_s_suppkey, sizeof(int32_t) * supplier_size);

    hipMemcpy(d_s_suppkey, s_suppkey, sizeof(int32_t) * supplier_size, hipMemcpyHostToDevice);

    int64_t *B2_supplier;
    int64_t *B2_nation;
    int64_t *B2_region;
    int64_t *B2_idx;
    hipMalloc(&B2_idx, sizeof(int64_t));
    hipMemset(B2_idx, 0, sizeof(int64_t));
    pipeline_7<<<std::ceil((float)supplier_size / (float)32), 32>>>(d_ps_suppkey, d_s_nationkey, d_s_suppkey, d_HT1_I, d_HT1_F, supplier_size, B1_region, B2_idx, B1_nation);

    int64_t h_B2_idx;
    hipMemcpy(&h_B2_idx, B2_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B2_idx, 0, sizeof(int64_t));
    hipMalloc(&B2_region, sizeof(int64_t) * h_B2_idx);
    hipMalloc(&B2_supplier, sizeof(int64_t) * h_B2_idx);
    hipMalloc(&B2_nation, sizeof(int64_t) * h_B2_idx);
    auto HT2 = cuco::static_map{h_B2_idx * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT2_F = HT2.ref(cuco::find);

    auto d_HT2_I = HT2.ref(cuco::insert);

    pipeline_6<<<std::ceil((float)supplier_size / (float)32), 32>>>(d_ps_suppkey, d_s_nationkey, d_s_suppkey, d_HT1_I, d_HT1_F, d_HT2_I, d_HT2_F, B2_idx, B2_supplier, B2_nation, supplier_size, B1_nation, B2_region, B1_region);

    double *d_ps_supplycost;

    hipMalloc(&d_ps_supplycost, sizeof(double) * partsupp_size);

    hipMemcpy(d_ps_supplycost, ps_supplycost, sizeof(double) * partsupp_size, hipMemcpyHostToDevice);

    int8_t *d_p_mfgr;

    hipMalloc(&d_p_mfgr, sizeof(int8_t) * part_size);

    hipMemcpy(d_p_mfgr, p_mfgr, sizeof(int8_t) * part_size, hipMemcpyHostToDevice);

    int32_t *d_agg_p_partkey;

    int8_t *d_agg_p_mfgr;

    double *d_min_supplycost;

    auto HT4 = cuco::static_map{partsupp_size * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT4_F = HT4.ref(cuco::find);

    auto d_HT4_I = HT4.ref(cuco::insert);

    pipeline_8<<<std::ceil((float)partsupp_size / (float)32), 32>>>(d_ps_suppkey, d_ps_partkey, d_HT2_I, d_HT2_F, d_HT3_I, d_HT3_F, d_HT4_I, d_HT4_F, B2_supplier, partsupp_size, B3_part, B2_region, B2_nation);

    auto HT4_size = HT4.size();

    hipMalloc(&d_agg_p_partkey, sizeof(int32_t) * HT4_size);

    hipMalloc(&d_agg_p_mfgr, sizeof(int8_t) * HT4_size);

    hipMalloc(&d_min_supplycost, sizeof(double) * HT4_size);

    hipMemset(d_agg_p_partkey, 0, sizeof(int32_t) * HT4_size);

    hipMemset(d_agg_p_mfgr, 0, sizeof(int8_t) * HT4_size);

    hipMemset(d_min_supplycost, 0, sizeof(double) * HT4_size);

    thrust::device_vector<int64_t> keys_4(HT4_size), vals_4(HT4_size);
    HT4.retrieve_all(keys_4.begin(), vals_4.begin());
    thrust::host_vector<int64_t> h_keys_4(HT4_size);
    thrust::copy(keys_4.begin(), keys_4.end(), h_keys_4.begin());
    thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_4(HT4_size);
    for (int i = 0; i < HT4_size; i++)
    {
        actual_dict_4[i] = cuco::make_pair(h_keys_4[i], i);
    }
    HT4.clear();
    HT4.insert(actual_dict_4.begin(), actual_dict_4.end());
    pipeline_9<<<std::ceil((float)partsupp_size / (float)32), 32>>>(d_ps_supplycost, d_ps_suppkey, d_ps_partkey, d_p_mfgr, d_p_partkey, d_agg_p_partkey, d_agg_p_mfgr, d_min_supplycost, d_HT2_I, d_HT2_F, d_HT3_I, d_HT3_F, d_HT4_I, d_HT4_F, B2_supplier, partsupp_size, B3_part, B2_region, B2_nation);

    size_t agg_size = HT4_size;

    std::cout << "Agg size 1: " << agg_size << std::endl;

    int64_t *B7_region;
    int64_t *B7_idx;
    hipMalloc(&B7_idx, sizeof(int64_t));
    hipMemset(B7_idx, 0, sizeof(int64_t));
    pipeline_11<<<std::ceil((float)region_size / (float)32), 32>>>(d_n_regionkey, d_r_name, d_r_regionkey, region_size, B7_idx);

    int64_t h_B7_idx;
    hipMemcpy(&h_B7_idx, B7_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B7_idx, 0, sizeof(int64_t));
    hipMalloc(&B7_region, sizeof(int64_t) * h_B7_idx);
    auto HT7 = cuco::static_map{h_B7_idx * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT7_F = HT7.ref(cuco::find);

    auto d_HT7_I = HT7.ref(cuco::insert);

    pipeline_10<<<std::ceil((float)region_size / (float)32), 32>>>(d_n_regionkey, d_r_name, d_r_regionkey, d_HT7_I, d_HT7_F, B7_region, region_size, B7_idx);

    int64_t *B8_region;
    int64_t *B8_nation;
    int64_t *B8_idx;
    hipMalloc(&B8_idx, sizeof(int64_t));
    hipMemset(B8_idx, 0, sizeof(int64_t));
    pipeline_13<<<std::ceil((float)nation_size / (float)32), 32>>>(d_s_nationkey, d_n_regionkey, d_n_nationkey, d_HT7_I, d_HT7_F, B8_idx, B7_region, nation_size);

    int64_t h_B8_idx;
    hipMemcpy(&h_B8_idx, B8_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B8_idx, 0, sizeof(int64_t));
    hipMalloc(&B8_region, sizeof(int64_t) * h_B8_idx);
    hipMalloc(&B8_nation, sizeof(int64_t) * h_B8_idx);
    auto HT8 = cuco::static_map{h_B8_idx * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT8_F = HT8.ref(cuco::find);

    auto d_HT8_I = HT8.ref(cuco::insert);

    pipeline_12<<<std::ceil((float)nation_size / (float)32), 32>>>(d_s_nationkey, d_n_regionkey, d_n_nationkey, d_HT8_I, d_HT8_F, d_HT7_I, d_HT7_F, B7_region, B8_region, B8_nation, B8_idx, nation_size);

    int64_t *B6_supplier;
    int64_t *B6_idx;
    hipMalloc(&B6_idx, sizeof(int64_t));
    hipMemset(B6_idx, 0, sizeof(int64_t));
    pipeline_15<<<std::ceil((float)supplier_size / (float)32), 32>>>(d_s_suppkey, d_ps_suppkey, supplier_size, B6_idx);

    int64_t h_B6_idx;
    hipMemcpy(&h_B6_idx, B6_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B6_idx, 0, sizeof(int64_t));
    hipMalloc(&B6_supplier, sizeof(int64_t) * h_B6_idx);
    auto HT6 = cuco::static_map{h_B6_idx * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT6_F = HT6.ref(cuco::find);

    auto d_HT6_I = HT6.ref(cuco::insert);

    pipeline_14<<<std::ceil((float)supplier_size / (float)32), 32>>>(d_s_suppkey, d_ps_suppkey, d_HT6_I, d_HT6_F, supplier_size, B6_supplier, B6_idx);

    int64_t *B5_agg;
    int64_t *B5_idx;
    hipMalloc(&B5_idx, sizeof(int64_t));
    hipMemset(B5_idx, 0, sizeof(int64_t));
    pipeline_17<<<std::ceil((float)agg_size / (float)32), 32>>>(d_ps_supplycost, d_ps_partkey, d_agg_p_partkey, d_min_supplycost, agg_size, B5_idx);

    int64_t h_B5_idx;
    hipMemcpy(&h_B5_idx, B5_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B5_idx, 0, sizeof(int64_t));
    hipMalloc(&B5_agg, sizeof(int64_t) * h_B5_idx);
    auto HT5 = cuco::static_map{h_B5_idx * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT5_F = HT5.ref(cuco::find);

    auto d_HT5_I = HT5.ref(cuco::insert);

    pipeline_16<<<std::ceil((float)agg_size / (float)32), 32>>>(d_ps_supplycost, d_ps_partkey, d_agg_p_partkey, d_min_supplycost, d_HT5_I, d_HT5_F, agg_size, B5_agg, B5_idx);

    double *d_mat_s_acctbal;

    int32_t *d_mat_p_partkey;

    int8_t *d_mat_p_mfgr;

    int8_t *d_mat_n_name;

    int64_t *d_mat_idx9;
    hipMalloc(&d_mat_idx9, sizeof(int64_t));

    hipMemset(d_mat_idx9, 0, sizeof(int64_t));

    pipeline_18<<<std::ceil((float)partsupp_size / (float)32), 32>>>(d_ps_supplycost, d_s_nationkey, d_ps_suppkey, d_ps_partkey, d_HT8_I, d_HT8_F, d_HT5_I, d_HT5_F, d_HT6_I, d_HT6_F, B5_agg, B6_supplier, B8_region, B8_nation, partsupp_size, d_mat_idx9);

    int64_t mat_size;

    hipMemcpy(&mat_size, d_mat_idx9, sizeof(int64_t), hipMemcpyDeviceToHost);

    std::cout << "mat size: " << mat_size << std::endl;

    hipMemset(d_mat_idx9, 0, sizeof(int64_t));

    hipMalloc(&d_mat_s_acctbal, sizeof(double) * mat_size);

    hipMalloc(&d_mat_p_partkey, sizeof(int32_t) * mat_size);

    hipMalloc(&d_mat_p_mfgr, sizeof(int8_t) * mat_size);

    hipMalloc(&d_mat_n_name, sizeof(int8_t) * mat_size);

    hipMemset(d_mat_s_acctbal, 0, sizeof(double) * mat_size);

    hipMemset(d_mat_p_partkey, 0, sizeof(int32_t) * mat_size);

    hipMemset(d_mat_p_mfgr, 0, sizeof(int8_t) * mat_size);

    hipMemset(d_mat_n_name, 0, sizeof(int8_t) * mat_size);
    double *d_s_acctbal;

    hipMalloc(&d_s_acctbal, sizeof(double) * supplier_size);

    hipMemcpy(d_s_acctbal, s_acctbal, sizeof(double) * supplier_size, hipMemcpyHostToDevice);

    int8_t *d_n_name;

    hipMalloc(&d_n_name, sizeof(int8_t) * nation_size);

    hipMemcpy(d_n_name, n_name, sizeof(int8_t) * nation_size, hipMemcpyHostToDevice);

    pipeline_19<<<std::ceil((float)partsupp_size / (float)32), 32>>>(d_s_acctbal, d_ps_suppkey, d_s_nationkey, d_ps_supplycost, d_n_name, d_agg_p_mfgr, d_agg_p_partkey, d_ps_partkey, d_mat_s_acctbal, d_mat_p_partkey, d_mat_p_mfgr, d_mat_n_name, d_HT8_I, d_HT8_F, d_HT5_I, d_HT5_F, d_HT6_I, d_HT6_F, B5_agg, B6_supplier, B8_region, B8_nation, partsupp_size, d_mat_idx9);

    double *p_mat_s_acctbal = (double *)malloc(sizeof(double) * mat_size);
    hipMemcpy(p_mat_s_acctbal, d_mat_s_acctbal, sizeof(double) * mat_size, hipMemcpyDeviceToHost);
    int8_t *p_mat_n_name = (int8_t *)malloc(sizeof(int8_t) * mat_size);
    hipMemcpy(p_mat_n_name, d_mat_n_name, sizeof(int8_t) * mat_size, hipMemcpyDeviceToHost);
    int32_t *p_mat_p_partkey = (int32_t *)malloc(sizeof(int32_t) * mat_size);
    hipMemcpy(p_mat_p_partkey, d_mat_p_partkey, sizeof(int32_t) * mat_size, hipMemcpyDeviceToHost);
    int8_t *p_mat_p_mfgr = (int8_t *)malloc(sizeof(int8_t) * mat_size);
    hipMemcpy(p_mat_p_mfgr, d_mat_p_mfgr, sizeof(int8_t) * mat_size, hipMemcpyDeviceToHost);
    for (int i = 0; i < mat_size; i++)
    {
        std::cout << p_mat_s_acctbal[i] << "\t";
        std::cout << (int)p_mat_n_name[i] << "\t";
        std::cout << p_mat_p_partkey[i] << "\t";
        std::cout << (int)p_mat_p_mfgr[i] << "\t";
        std::cout << std::endl;
    }
}

int main(int argc, const char **argv)
{
    std::string dbDir = getDataDir(argv, argc);
    std::string nation_file = dbDir + "nation.parquet";
    std::string part_file = dbDir + "part.parquet";
    std::string partsupp_file = dbDir + "partsupp.parquet";
    std::string supplier_file = dbDir + "supplier.parquet";
    std::string region_file = dbDir + "region.parquet";

    // auto lineitem_table  = getArrowTable(lineitem_file);
    auto nation_table = getArrowTable(nation_file);
    auto region_table = getArrowTable(region_file);
    auto part_table = getArrowTable(part_file);
    auto partsupp_table = getArrowTable(partsupp_file);
    auto supplier_table = getArrowTable(supplier_file);
    size_t nation_size = nation_table->num_rows();
    size_t part_size = part_table->num_rows();
    size_t partsupp_size = partsupp_table->num_rows();
    size_t region_size = region_table->num_rows();
    size_t supplier_size = supplier_table->num_rows();

    auto n_nationkey = read_column_typecasted<int32_t>(nation_table, "n_nationkey");
    auto n_regionkey = read_column_typecasted<int32_t>(nation_table, "n_regionkey");
    auto n_name = read_string_dict_encoded_column(nation_table, "n_name");

    auto p_partkey = read_column_typecasted<int32_t>(part_table, "p_partkey");
    auto p_mfgr = read_string_dict_encoded_column(part_table, "p_mfgr");
    auto p_size = read_column_typecasted<int32_t>(part_table, "p_size");
    auto p_type = read_string_column(part_table, "p_type");

    auto ps_partkey = read_column_typecasted<int32_t>(partsupp_table, "ps_partkey");
    auto ps_suppkey = read_column_typecasted<int32_t>(partsupp_table, "ps_suppkey");
    auto ps_supplycost = read_column<double>(partsupp_table, "ps_supplycost");

    auto r_name = read_string_dict_encoded_column(region_table, "r_name");
    auto r_regionkey = read_column_typecasted<int32_t>(region_table, "r_regionkey");

    auto s_nationkey = read_column_typecasted<int32_t>(supplier_table, "s_nationkey");
    auto s_suppkey = read_column_typecasted<int32_t>(supplier_table, "s_suppkey");
    auto s_acctbal = read_column<double>(supplier_table, "s_acctbal");

    // for (auto p: c_mktsegment->dict) {
    //   std::cout << p.first << " " << (int)p.second << std::endl;
    // }
    control(
        n_name->column,
        n_nationkey.data(),
        n_regionkey.data(),
        p_mfgr->column,
        p_partkey.data(),
        p_size.data(),
        p_type,
        ps_partkey.data(),
        ps_suppkey.data(),
        ps_supplycost.data(),
        r_name->column,
        r_regionkey.data(),
        s_nationkey.data(),
        s_acctbal.data(),
        s_suppkey.data(),
        nation_size,
        part_size,
        partsupp_size,
        region_size,
        supplier_size);
}