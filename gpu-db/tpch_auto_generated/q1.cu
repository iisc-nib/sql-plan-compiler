#include "hip/hip_runtime.h"
#include "utils.h"

#include <cuco/static_map.cuh>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
namespace cg = cooperative_groups;

template <typename TY_HT0_I, typename TY_HT0_F>
__global__ void pipeline_0(double *l_discount, int32_t *l_shipdate, int8_t *l_linestatus, int8_t *l_returnflag, double *l_extendedprice, double *l_tax, TY_HT0_I HT0_I, TY_HT0_F HT0_F, size_t lineitem_size)
{
  int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid >= lineitem_size)
    return;
  int32_t reg_l_shipdate = l_shipdate[tid];
  if (!(reg_l_shipdate <= 10471))
    return;
  double reg_l_extendedprice = l_extendedprice[tid];
  double reg_l_discount = l_discount[tid];
  double reg_sum_disc_price = (reg_l_extendedprice * (1 - reg_l_discount));
  double reg_l_tax = l_tax[tid];
  double reg_sum_charge = ((reg_l_extendedprice * (1 - reg_l_discount)) * (1 + reg_l_tax));
  int8_t reg_l_returnflag = l_returnflag[tid];
  int8_t reg_l_linestatus = l_linestatus[tid];
  int64_t key0 = 0;
  key0 |= (((int64_t)reg_l_returnflag) << 0);
  key0 |= (((int64_t)reg_l_linestatus) << 8);
  auto thread = cg::tiled_partition<1>(cg::this_thread_block());
  HT0_I.insert(thread, cuco::pair{key0, 1});
}

template <typename TY_HT0_I, typename TY_HT0_F>
__global__ void pipeline_1(double *l_discount, int32_t *l_shipdate, int8_t *l_linestatus, int8_t *l_returnflag, double *l_extendedprice, double *l_tax, int64_t *l_quantity, double *sum_disc_price, int8_t *agg_l_linestatus, double *sum_discount, int64_t *sum_qty, double *sum_charge, double *sum_base_price, int64_t *count_order, int8_t *agg_l_returnflag, TY_HT0_I HT0_I, TY_HT0_F HT0_F, size_t lineitem_size)
{
  int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid >= lineitem_size)
    return;
  int32_t reg_l_shipdate = l_shipdate[tid];
  if (!(reg_l_shipdate <= 10471))
    return;
  double reg_l_extendedprice = l_extendedprice[tid];
  double reg_l_discount = l_discount[tid];
  double reg_sum_disc_price = (reg_l_extendedprice * (1 - reg_l_discount));
  double reg_l_tax = l_tax[tid];
  double reg_sum_charge = ((reg_l_extendedprice * (1 - reg_l_discount)) * (1 + reg_l_tax));
  int8_t reg_l_returnflag = l_returnflag[tid];
  int8_t reg_l_linestatus = l_linestatus[tid];
  int64_t key0 = 0;
  key0 |= (((int64_t)reg_l_returnflag) << 0);
  key0 |= (((int64_t)reg_l_linestatus) << 8);
  auto slot0 = HT0_F.find(key0);
  int64_t reg_l_quantity = l_quantity[tid];
  agg_l_returnflag[slot0->second] = reg_l_returnflag;
  agg_l_linestatus[slot0->second] = reg_l_linestatus;
  aggregate_sum(&(sum_qty[slot0->second]), reg_l_quantity);
  aggregate_sum(&(sum_discount[slot0->second]), reg_l_discount);
  aggregate_sum(&(sum_base_price[slot0->second]), reg_l_extendedprice);
  aggregate_sum(&(sum_disc_price[slot0->second]), reg_sum_disc_price);
  aggregate_sum(&(sum_charge[slot0->second]), reg_sum_charge);
  aggregate_sum(&(count_order[slot0->second]), 1);
}

void control(
    double *l_discount,
    double *l_extendedprice,
    int8_t *l_linestatus,
    int64_t *l_quantity,
    int8_t *l_returnflag,
    int32_t *l_shipdate,
    double *l_tax,
    size_t lineitem_size)
{
  double *d_l_discount;

  hipMalloc(&d_l_discount, sizeof(double) * lineitem_size);

  hipMemcpy(d_l_discount, l_discount, sizeof(double) * lineitem_size, hipMemcpyHostToDevice);

  int32_t *d_l_shipdate;

  hipMalloc(&d_l_shipdate, sizeof(int32_t) * lineitem_size);

  hipMemcpy(d_l_shipdate, l_shipdate, sizeof(int32_t) * lineitem_size, hipMemcpyHostToDevice);

  int8_t *d_l_linestatus;

  hipMalloc(&d_l_linestatus, sizeof(int8_t) * lineitem_size);

  hipMemcpy(d_l_linestatus, l_linestatus, sizeof(int8_t) * lineitem_size, hipMemcpyHostToDevice);

  int8_t *d_l_returnflag;

  hipMalloc(&d_l_returnflag, sizeof(int8_t) * lineitem_size);

  hipMemcpy(d_l_returnflag, l_returnflag, sizeof(int8_t) * lineitem_size, hipMemcpyHostToDevice);

  double *d_l_extendedprice;

  hipMalloc(&d_l_extendedprice, sizeof(double) * lineitem_size);

  hipMemcpy(d_l_extendedprice, l_extendedprice, sizeof(double) * lineitem_size, hipMemcpyHostToDevice);

  double *d_l_tax;

  hipMalloc(&d_l_tax, sizeof(double) * lineitem_size);

  hipMemcpy(d_l_tax, l_tax, sizeof(double) * lineitem_size, hipMemcpyHostToDevice);

  int64_t *d_l_quantity;

  hipMalloc(&d_l_quantity, sizeof(int64_t) * lineitem_size);

  hipMemcpy(d_l_quantity, l_quantity, sizeof(int64_t) * lineitem_size, hipMemcpyHostToDevice);

  double *d_sum_disc_price;

  int8_t *d_agg_l_linestatus;

  double *d_sum_discount;

  int64_t *d_sum_qty;

  double *d_sum_charge;

  double *d_sum_base_price;

  int64_t *d_count_order;

  int8_t *d_agg_l_returnflag;

  auto HT0 = cuco::static_map{lineitem_size * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

  auto d_HT0_F = HT0.ref(cuco::find);

  auto d_HT0_I = HT0.ref(cuco::insert);

  pipeline_0<<<std::ceil((float)lineitem_size / (float)32), 32>>>(d_l_discount, d_l_shipdate, d_l_linestatus, d_l_returnflag, d_l_extendedprice, d_l_tax, d_HT0_I, d_HT0_F, lineitem_size);

  auto HT0_size = HT0.size();

  hipMalloc(&d_sum_disc_price, sizeof(double) * HT0_size);

  hipMalloc(&d_agg_l_linestatus, sizeof(int8_t) * HT0_size);

  hipMalloc(&d_sum_discount, sizeof(double) * HT0_size);

  hipMalloc(&d_sum_qty, sizeof(int64_t) * HT0_size);

  hipMalloc(&d_sum_charge, sizeof(double) * HT0_size);

  hipMalloc(&d_sum_base_price, sizeof(double) * HT0_size);

  hipMalloc(&d_count_order, sizeof(int64_t) * HT0_size);

  hipMalloc(&d_agg_l_returnflag, sizeof(int8_t) * HT0_size);

  hipMemset(d_sum_disc_price, 0, sizeof(double) * HT0_size);

  hipMemset(d_agg_l_linestatus, 0, sizeof(int8_t) * HT0_size);

  hipMemset(d_sum_discount, 0, sizeof(double) * HT0_size);

  hipMemset(d_sum_qty, 0, sizeof(int64_t) * HT0_size);

  hipMemset(d_sum_charge, 0, sizeof(double) * HT0_size);

  hipMemset(d_sum_base_price, 0, sizeof(double) * HT0_size);

  hipMemset(d_count_order, 0, sizeof(int64_t) * HT0_size);

  hipMemset(d_agg_l_returnflag, 0, sizeof(int8_t) * HT0_size);

  thrust::device_vector<int64_t> keys_0(HT0_size), vals_0(HT0_size);
  HT0.retrieve_all(keys_0.begin(), vals_0.begin());
  thrust::host_vector<int64_t> h_keys_0(HT0_size);
  thrust::copy(keys_0.begin(), keys_0.end(), h_keys_0.begin());
  thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_0(HT0_size);
  for (int i = 0; i < HT0_size; i++)
  {
    actual_dict_0[i] = cuco::make_pair(h_keys_0[i], i);
  }
  HT0.clear();
  HT0.insert(actual_dict_0.begin(), actual_dict_0.end());
  pipeline_1<<<std::ceil((float)lineitem_size / (float)32), 32>>>(d_l_discount, d_l_shipdate, d_l_linestatus, d_l_returnflag, d_l_extendedprice, d_l_tax, d_l_quantity, d_sum_disc_price, d_agg_l_linestatus, d_sum_discount, d_sum_qty, d_sum_charge, d_sum_base_price, d_count_order, d_agg_l_returnflag, d_HT0_I, d_HT0_F, lineitem_size);

  size_t agg_size = HT0_size;
  int8_t *p_agg_l_returnflag = (int8_t *)malloc(sizeof(int8_t) * agg_size);
  hipMemcpy(p_agg_l_returnflag, d_agg_l_returnflag, sizeof(int8_t) * agg_size, hipMemcpyDeviceToHost);
  int8_t *p_agg_l_linestatus = (int8_t *)malloc(sizeof(int8_t) * agg_size);
  hipMemcpy(p_agg_l_linestatus, d_agg_l_linestatus, sizeof(int8_t) * agg_size, hipMemcpyDeviceToHost);
  int64_t *p_sum_qty = (int64_t *)malloc(sizeof(int64_t) * agg_size);
  hipMemcpy(p_sum_qty, d_sum_qty, sizeof(int64_t) * agg_size, hipMemcpyDeviceToHost);
  double *p_sum_discount = (double *)malloc(sizeof(double) * agg_size);
  hipMemcpy(p_sum_discount, d_sum_discount, sizeof(double) * agg_size, hipMemcpyDeviceToHost);
  double *p_sum_base_price = (double *)malloc(sizeof(double) * agg_size);
  hipMemcpy(p_sum_base_price, d_sum_base_price, sizeof(double) * agg_size, hipMemcpyDeviceToHost);
  double *p_sum_disc_price = (double *)malloc(sizeof(double) * agg_size);
  hipMemcpy(p_sum_disc_price, d_sum_disc_price, sizeof(double) * agg_size, hipMemcpyDeviceToHost);
  double *p_sum_charge = (double *)malloc(sizeof(double) * agg_size);
  hipMemcpy(p_sum_charge, d_sum_charge, sizeof(double) * agg_size, hipMemcpyDeviceToHost);
  int64_t *p_count_order = (int64_t *)malloc(sizeof(int64_t) * agg_size);
  hipMemcpy(p_count_order, d_count_order, sizeof(int64_t) * agg_size, hipMemcpyDeviceToHost);
  for (int i = 0; i < agg_size; i++)
  {
    std::cout << (int)p_agg_l_returnflag[i] << "\t";
    std::cout << p_agg_l_returnflag[i] << "\t";
    std::cout << (int)p_agg_l_linestatus[i] << "\t";
    std::cout << p_agg_l_linestatus[i] << "\t";
    std::cout << p_sum_qty[i] << "\t";
    std::cout << p_sum_discount[i] << "\t";
    std::cout << p_sum_base_price[i] << "\t";
    std::cout << p_sum_disc_price[i] << "\t";
    std::cout << p_sum_charge[i] << "\t";
    std::cout << p_count_order[i] << "\t";
    std::cout << std::endl;
  }
}
int main(int argc, const char **argv)
{
  std::string dbDir = getDataDir(argv, argc);
  std::string lineitem_file = dbDir + "lineitem.parquet";

  auto lineitem_table = getArrowTable(lineitem_file);
  size_t lineitem_size = lineitem_table->num_rows();

  auto l_quantity = read_column<int64_t>(lineitem_table, "l_quantity");
  auto l_shipdate = read_column<int32_t>(lineitem_table, "l_shipdate");
  auto l_extendedprice = read_column<double>(lineitem_table, "l_extendedprice");
  auto l_discount = read_column<double>(lineitem_table, "l_discount");
  auto l_tax = read_column<double>(lineitem_table, "l_tax");
  StringDictEncodedColumn *l_returnflag =
      read_string_dict_encoded_column(lineitem_table, "l_returnflag");
  StringDictEncodedColumn *l_linestatus =
      read_string_dict_encoded_column(lineitem_table, "l_linestatus");
  control(l_discount.data(),
          l_extendedprice.data(),
          l_linestatus->column,
          l_quantity.data(),
          l_returnflag->column,
          l_shipdate.data(),
          l_tax.data(),
          lineitem_size);
}