#include "hip/hip_runtime.h"

#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <cuco/static_map.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
__global__ void count_pipeline_55c975d43320(DBStringType *customer__c_mktsegment,
int * BUF_IDX_55c975d38d60,
size_t customer_size
) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg__customer__c_mktsegment = customer__c_mktsegment[tid];
if (!(evaluatePredicate(reg__customer__c_mktsegment, "BUILDING", Predicate::eq))) return;
atomicAdd(BUF_IDX_55c975d38d60, 1);
return;
}
template<typename HASHTABLE_INSERT>
__global__ void main_pipeline_55c975d43320(DBI32Type *customer__c_custkey,
DBStringType *customer__c_mktsegment,
uint64_t * BUF_55c975d38d60,
int * BUF_IDX_55c975d38d60,
HASHTABLE_INSERT HT_55c975d38d60,
size_t customer_size
) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg__customer__c_mktsegment = customer__c_mktsegment[tid];
if (!(evaluatePredicate(reg__customer__c_mktsegment, "BUILDING", Predicate::eq))) return;
auto reg__customer__c_custkey = customer__c_custkey[tid];
int64_t KEY_55c975d38d60 = 0;
KEY_55c975d38d60 <<= 32;
KEY_55c975d38d60  |= reg__customer__c_custkey;

auto buf_idx_55c975d38d60 = atomicAdd(BUF_IDX_55c975d38d60, 1);
HT_55c975d38d60.insert(cuco::pair{KEY_55c975d38d60, buf_idx_55c975d38d60});
BUF_55c975d38d60[buf_idx_55c975d38d60 * 1 + 0] = tid;
return;
}
template<typename HASHTABLE_FIND>
__global__ void count_pipeline_55c975d43e60(DBI32Type *orders__o_custkey,
DBDateType *orders__o_orderdate,
uint64_t* BUF_55c975d38d60,
int * BUF_IDX_55c975d38ec0,
HASHTABLE_FIND HT_55c975d38d60,
size_t orders_size
) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg__orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg__orders__o_orderdate, 9204, Predicate::lt))) return;
auto reg__orders__o_custkey = orders__o_custkey[tid];
int64_t KEY_55c975d38d60 = 0;
KEY_55c975d38d60 <<= 32;
KEY_55c975d38d60  |= reg__orders__o_custkey;

auto SLOT_55c975d38d60 = HT_55c975d38d60.find(KEY_55c975d38d60);
if (SLOT_55c975d38d60 == HT_55c975d38d60.end()) return;
auto buf_idx_55c975d38d60 = SLOT_55c975d38d60->second;
atomicAdd(BUF_IDX_55c975d38ec0, 1);
return;
}
template<typename HASHTABLE_FIND, typename HASHTABLE_INSERT>
__global__ void main_pipeline_55c975d43e60(DBI32Type *orders__o_custkey,
DBDateType *orders__o_orderdate,
DBI32Type *orders__o_orderkey,
uint64_t* BUF_55c975d38d60,
uint64_t * BUF_55c975d38ec0,
int * BUF_IDX_55c975d38ec0,
HASHTABLE_FIND HT_55c975d38d60,
HASHTABLE_INSERT HT_55c975d38ec0,
size_t orders_size
) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg__orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg__orders__o_orderdate, 9204, Predicate::lt))) return;
auto reg__orders__o_custkey = orders__o_custkey[tid];
int64_t KEY_55c975d38d60 = 0;
KEY_55c975d38d60 <<= 32;
KEY_55c975d38d60  |= reg__orders__o_custkey;

auto SLOT_55c975d38d60 = HT_55c975d38d60.find(KEY_55c975d38d60);
if (SLOT_55c975d38d60 == HT_55c975d38d60.end()) return;
auto buf_idx_55c975d38d60 = SLOT_55c975d38d60->second;
auto reg__orders__o_orderkey = orders__o_orderkey[tid];
int64_t KEY_55c975d38ec0 = 0;
KEY_55c975d38ec0 <<= 32;
KEY_55c975d38ec0  |= reg__orders__o_orderkey;

auto buf_idx_55c975d38ec0 = atomicAdd(BUF_IDX_55c975d38ec0, 1);
HT_55c975d38ec0.insert(cuco::pair{KEY_55c975d38ec0, buf_idx_55c975d38ec0});
BUF_55c975d38ec0[buf_idx_55c975d38ec0 * 2 + 0] = BUF_55c975d38d60[buf_idx_55c975d38d60*1 + 0];
BUF_55c975d38ec0[buf_idx_55c975d38ec0 * 2 + 1] = tid;
return;
}
template<typename HASHTABLE_FIND, typename HASHTABLE_INSERT>
__global__ void count_pipeline_55c975d45c00(DBDecimalType *lineitem__l_discount,
DBDecimalType *lineitem__l_extendedprice,
DBI32Type *lineitem__l_orderkey,
DBDateType *lineitem__l_shipdate,
uint64_t* BUF_55c975d38ec0,
HASHTABLE_INSERT HT_55c975cf4cf0,
HASHTABLE_FIND HT_55c975d38ec0,
size_t lineitem_size
) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg__lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg__lineitem__l_shipdate, 9204, Predicate::gt))) return;
auto reg__lineitem__l_orderkey = lineitem__l_orderkey[tid];
int64_t KEY_55c975d38ec0 = 0;
KEY_55c975d38ec0 <<= 32;
KEY_55c975d38ec0  |= reg__lineitem__l_orderkey;

auto SLOT_55c975d38ec0 = HT_55c975d38ec0.find(KEY_55c975d38ec0);
if (SLOT_55c975d38ec0 == HT_55c975d38ec0.end()) return;
auto buf_idx_55c975d38ec0 = SLOT_55c975d38ec0->second;
auto reg__lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg__lineitem__l_discount = lineitem__l_discount[tid];
int64_t KEY_55c975cf4cf0 = 0;
KEY_55c975cf4cf0 <<= 32;
KEY_55c975cf4cf0  |= reg__lineitem__l_orderkey;

HT_55c975cf4cf0.insert(cuco::pair{KEY_55c975cf4cf0, 1});
return;
}
template<typename HASHTABLE_FIND>
__global__ void main_pipeline_55c975d45c00(DBI32Type *KEY_55c975cf4cf0lineitem__l_orderkey,
DBDecimalType *aggr0__tmp_attr0,
DBDateType *aggr__o_orderdate,
DBI32Type *aggr__o_shippriority,
DBDecimalType *lineitem__l_discount,
DBDecimalType *lineitem__l_extendedprice,
DBI32Type *lineitem__l_orderkey,
DBDateType *lineitem__l_shipdate,
DBDateType *orders__o_orderdate,
DBI32Type *orders__o_shippriority,
uint64_t* BUF_55c975d38ec0,
HASHTABLE_FIND HT_55c975cf4cf0,
HASHTABLE_FIND HT_55c975d38ec0,
size_t lineitem_size
) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg__lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg__lineitem__l_shipdate, 9204, Predicate::gt))) return;
auto reg__lineitem__l_orderkey = lineitem__l_orderkey[tid];
int64_t KEY_55c975d38ec0 = 0;
KEY_55c975d38ec0 <<= 32;
KEY_55c975d38ec0  |= reg__lineitem__l_orderkey;

auto SLOT_55c975d38ec0 = HT_55c975d38ec0.find(KEY_55c975d38ec0);
if (SLOT_55c975d38ec0 == HT_55c975d38ec0.end()) return;
auto buf_idx_55c975d38ec0 = SLOT_55c975d38ec0->second;
auto reg__lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg__lineitem__l_discount = lineitem__l_discount[tid];
int64_t KEY_55c975cf4cf0 = 0;
KEY_55c975cf4cf0 <<= 32;
KEY_55c975cf4cf0  |= reg__lineitem__l_orderkey;

auto buf_idx_55c975cf4cf0 = HT_55c975cf4cf0.find(KEY_55c975cf4cf0)->second;
KEY_55c975cf4cf0lineitem__l_orderkey[buf_idx_55c975cf4cf0] = reg__lineitem__l_orderkey;
auto reg__map0__tmp_attr1 = (reg__lineitem__l_extendedprice) * ((1) - (reg__lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_55c975cf4cf0], reg__map0__tmp_attr1);
auto reg__orders__o_shippriority = orders__o_shippriority[BUF_55c975d38ec0[buf_idx_55c975d38ec0*2 + 1]];
aggregate_any(&aggr__o_shippriority[buf_idx_55c975cf4cf0], reg__orders__o_shippriority);
auto reg__orders__o_orderdate = orders__o_orderdate[BUF_55c975d38ec0[buf_idx_55c975d38ec0*2 + 1]];
aggregate_any(&aggr__o_orderdate[buf_idx_55c975cf4cf0], reg__orders__o_orderdate);
return;
}
__global__ void count_pipeline_55c975d4fa00(size_t MAT_55c975cf4cf0_size
) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= MAT_55c975cf4cf0_size) return;
}
__global__ void main_pipeline_55c975d4fa00(DBDecimalType *MAT_55c975cd41b0aggr0__tmp_attr0,
DBDateType *MAT_55c975cd41b0aggr__o_orderdate,
DBI32Type *MAT_55c975cd41b0aggr__o_shippriority,
DBI32Type *MAT_55c975cd41b0lineitem__l_orderkey,
DBDecimalType *aggr0__tmp_attr0,
DBDateType *aggr__o_orderdate,
DBI32Type *aggr__o_shippriority,
DBI32Type *lineitem__l_orderkey,
size_t MAT_55c975cf4cf0_size
) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= MAT_55c975cf4cf0_size) return;
auto reg__lineitem__l_orderkey = lineitem__l_orderkey[tid];
MAT_55c975cd41b0lineitem__l_orderkey[tid] = reg__lineitem__l_orderkey;
auto reg__aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT_55c975cd41b0aggr0__tmp_attr0[tid] = reg__aggr0__tmp_attr0;
auto reg__aggr__o_orderdate = aggr__o_orderdate[tid];
MAT_55c975cd41b0aggr__o_orderdate[tid] = reg__aggr__o_orderdate;
auto reg__aggr__o_shippriority = aggr__o_shippriority[tid];
MAT_55c975cd41b0aggr__o_shippriority[tid] = reg__aggr__o_shippriority;
}

extern "C" void control(
    DBI32Type* d_nation__n_nationkey,
    DBStringType* d_nation__n_name,
    DBI32Type* d_nation__n_regionkey,
    DBStringType* d_nation__n_comment,
    size_t nation_size,
    DBI32Type* d_supplier__s_suppkey,
    DBI32Type* d_supplier__s_nationkey,
    DBStringType* d_supplier__s_name,
    DBStringType* d_supplier__s_address,
    DBStringType* d_supplier__s_phone,
    DBDecimalType* d_supplier__s_acctbal,
    DBStringType* d_supplier__s_comment,
    size_t supplier_size,
    DBI32Type* d_partsupp__ps_suppkey,
    DBI32Type* d_partsupp__ps_partkey,
    DBI32Type* d_partsupp__ps_availqty,
    DBDecimalType* d_partsupp__ps_supplycost,
    DBStringType* d_partsupp__ps_comment,
    size_t partsupp_size,
    DBI32Type* d_part__p_partkey,
    DBStringType* d_part__p_name,
    DBStringType* d_part__p_mfgr,
    DBStringType* d_part__p_brand,
    DBStringType* d_part__p_type,
    DBI32Type* d_part__p_size,
    DBStringType* d_part__p_container,
    DBDecimalType* d_part__p_retailprice,
    DBStringType* d_part__p_comment,
    size_t part_size,
    DBI32Type* d_lineitem__l_orderkey,
    DBI32Type* d_lineitem__l_partkey,
    DBI32Type* d_lineitem__l_suppkey,
    DBI64Type* d_lineitem__l_linenumber,
    DBDecimalType* d_lineitem__l_quantity,
    DBDecimalType* d_lineitem__l_extendedprice,
    DBDecimalType* d_lineitem__l_discount,
    DBDecimalType* d_lineitem__l_tax,
    DBCharType* d_lineitem__l_returnflag,
    DBCharType* d_lineitem__l_linestatus,
    DBI32Type* d_lineitem__l_shipdate,
    DBI32Type* d_lineitem__l_commitdate,
    DBI32Type* d_lineitem__l_receiptdate,
    DBStringType* d_lineitem__l_shipinstruct,
    DBStringType* d_lineitem__l_shipmode,
    DBStringType* d_lineitem__comments,
    size_t lineitem_size,
    DBI32Type* d_orders__o_orderkey,
    DBCharType* d_orders__o_orderstatus,
    DBI32Type* d_orders__o_custkey,
    DBDecimalType* d_orders__o_totalprice,
    DBI32Type* d_orders__o_orderdate,
    DBStringType* d_orders__o_orderpriority,
    DBStringType* d_orders__o_clerk,
    DBI32Type* d_orders__o_shippriority,
    DBStringType* d_orders__o_comment,
    size_t orders_size,
    DBI32Type* d_customer__c_custkey,
    DBStringType* d_customer__c_name,
    DBStringType* d_customer__c_address,
    DBI32Type* d_customer__c_nationkey,
    DBStringType* d_customer__c_phone,
    DBDecimalType* d_customer__c_acctbal,
    DBStringType* d_customer__c_mktsegment,
    DBStringType* d_customer__c_comment,
    size_t customer_size,
    DBI32Type* d_region__r_regionkey,
    DBStringType* d_region__r_name,
    DBStringType* d_region__r_comment,
    size_t region_size
    )
{
int *d_BUF_IDX_55c975d38d60;
hipMalloc(&d_BUF_IDX_55c975d38d60, sizeof(int));
hipMemset(d_BUF_IDX_55c975d38d60,0 , sizeof(int));
count_pipeline_55c975d43320<<<std::ceil((float)customer_size/(float)32), 32>>>(d_customer__c_mktsegment, d_BUF_IDX_55c975d38d60, customer_size);

int BUF_IDX_55c975d38d60;
hipMemcpy(&BUF_IDX_55c975d38d60, d_BUF_IDX_55c975d38d60, sizeof(int), hipMemcpyDeviceToHost);
std::cout << "first build, size: " << BUF_IDX_55c975d38d60 << std::endl;
hipMemset(d_BUF_IDX_55c975d38d60,0 , sizeof(int));
auto HT_55c975d38d60 = cuco::static_map{ BUF_IDX_55c975d38d60* 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};
uint64_t *d_BUF_55c975d38d60;
hipMalloc(&d_BUF_55c975d38d60, sizeof(uint64_t) * 1 * BUF_IDX_55c975d38d60);
hipMemset(d_BUF_55c975d38d60,0 , sizeof(uint64_t) * 1 * BUF_IDX_55c975d38d60);
main_pipeline_55c975d43320<<<std::ceil((float)customer_size/(float)32), 32>>>(d_customer__c_custkey, d_customer__c_mktsegment, d_BUF_55c975d38d60, d_BUF_IDX_55c975d38d60, HT_55c975d38d60.ref(cuco::insert), customer_size);
std::cout << "customer hash table size: " << HT_55c975d38d60.size() << std::endl;
int *d_BUF_IDX_55c975d38ec0;
hipMalloc(&d_BUF_IDX_55c975d38ec0, sizeof(int));
hipMemset(d_BUF_IDX_55c975d38ec0,0 , sizeof(int));
count_pipeline_55c975d43e60<<<std::ceil((float)orders_size/(float)32), 32>>>(d_orders__o_custkey, d_orders__o_orderdate, d_BUF_55c975d38d60, d_BUF_IDX_55c975d38ec0, HT_55c975d38d60.ref(cuco::find), orders_size);

int BUF_IDX_55c975d38ec0;
hipMemcpy(&BUF_IDX_55c975d38ec0, d_BUF_IDX_55c975d38ec0, sizeof(int), hipMemcpyDeviceToHost);
std::cout << "second build, size: " << BUF_IDX_55c975d38ec0 << std::endl;
hipMemset(d_BUF_IDX_55c975d38ec0,0 , sizeof(int));
auto HT_55c975d38ec0 = cuco::static_map{ BUF_IDX_55c975d38ec0* 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};
uint64_t *d_BUF_55c975d38ec0;
hipMalloc(&d_BUF_55c975d38ec0, sizeof(uint64_t) * 2 * BUF_IDX_55c975d38ec0);
hipMemset(d_BUF_55c975d38ec0,0 , sizeof(uint64_t) * 2 * BUF_IDX_55c975d38ec0);
main_pipeline_55c975d43e60<<<std::ceil((float)orders_size/(float)32), 32>>>(d_orders__o_custkey, d_orders__o_orderdate, d_orders__o_orderkey, d_BUF_55c975d38d60, d_BUF_55c975d38ec0, d_BUF_IDX_55c975d38ec0, HT_55c975d38d60.ref(cuco::find), HT_55c975d38ec0.ref(cuco::insert), orders_size);



auto HT_55c975cf4cf0 = cuco::static_map{ 35555* 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};
count_pipeline_55c975d45c00<<<std::ceil((float)lineitem_size/(float)32), 32>>>(d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_BUF_55c975d38ec0, HT_55c975cf4cf0.ref(cuco::insert), HT_55c975d38ec0.ref(cuco::find), lineitem_size);
std::cout << "Aggregation size: " << HT_55c975cf4cf0.size() << std::endl;


thrust::device_vector<int64_t> keys_55c975cf4cf0(HT_55c975cf4cf0.size()), vals_55c975cf4cf0(HT_55c975cf4cf0.size());
HT_55c975cf4cf0.retrieve_all(keys_55c975cf4cf0.begin(), vals_55c975cf4cf0.begin());
thrust::host_vector<int64_t> h_keys_55c975cf4cf0(HT_55c975cf4cf0.size());
thrust::copy(keys_55c975cf4cf0.begin(), keys_55c975cf4cf0.end(), h_keys_55c975cf4cf0.begin());
thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_55c975cf4cf0(HT_55c975cf4cf0.size());
for (int i=0; i < HT_55c975cf4cf0.size(); i++) {{
actual_dict_55c975cf4cf0[i] = cuco::make_pair(h_keys_55c975cf4cf0[i], i);
}}
HT_55c975cf4cf0.clear();
HT_55c975cf4cf0.insert(actual_dict_55c975cf4cf0.begin(), actual_dict_55c975cf4cf0.end());
DBDecimalType*  d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * HT_55c975cf4cf0.size());
hipMemset(d_aggr0__tmp_attr0,0 , sizeof(DBDecimalType) * HT_55c975cf4cf0.size());
DBI32Type*  d_aggr__o_shippriority;
hipMalloc(&d_aggr__o_shippriority, sizeof(DBI32Type) * HT_55c975cf4cf0.size());
hipMemset(d_aggr__o_shippriority,0 , sizeof(DBI32Type) * HT_55c975cf4cf0.size());
DBDateType*  d_aggr__o_orderdate;
hipMalloc(&d_aggr__o_orderdate, sizeof(DBDateType) * HT_55c975cf4cf0.size());
hipMemset(d_aggr__o_orderdate,0 , sizeof(DBDateType) * HT_55c975cf4cf0.size());
auto MAT_55c975cf4cf0_size = HT_55c975cf4cf0.size();
DBI32Type*  d_KEY_55c975cf4cf0lineitem__l_orderkey;
hipMalloc(&d_KEY_55c975cf4cf0lineitem__l_orderkey, sizeof(DBI32Type) * HT_55c975cf4cf0.size());
hipMemset(d_KEY_55c975cf4cf0lineitem__l_orderkey,0 , sizeof(DBI32Type) * HT_55c975cf4cf0.size());
main_pipeline_55c975d45c00<<<std::ceil((float)lineitem_size/(float)32), 32>>>(d_KEY_55c975cf4cf0lineitem__l_orderkey, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_orders__o_orderdate, d_orders__o_shippriority, d_BUF_55c975d38ec0, HT_55c975cf4cf0.ref(cuco::find), HT_55c975d38ec0.ref(cuco::find), lineitem_size);


auto MAT_55c975cd41b0lineitem__l_orderkey = (DBI32Type*)malloc(sizeof(DBI32Type) * MAT_55c975cf4cf0_size);
DBI32Type* d_MAT_55c975cd41b0lineitem__l_orderkey;
hipMalloc(&d_MAT_55c975cd41b0lineitem__l_orderkey, sizeof(DBI32Type) * MAT_55c975cf4cf0_size);
auto MAT_55c975cd41b0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * MAT_55c975cf4cf0_size);
DBDecimalType* d_MAT_55c975cd41b0aggr0__tmp_attr0;
hipMalloc(&d_MAT_55c975cd41b0aggr0__tmp_attr0, sizeof(DBDecimalType) * MAT_55c975cf4cf0_size);
auto MAT_55c975cd41b0aggr__o_orderdate = (DBDateType*)malloc(sizeof(DBDateType) * MAT_55c975cf4cf0_size);
DBDateType* d_MAT_55c975cd41b0aggr__o_orderdate;
hipMalloc(&d_MAT_55c975cd41b0aggr__o_orderdate, sizeof(DBDateType) * MAT_55c975cf4cf0_size);
auto MAT_55c975cd41b0aggr__o_shippriority = (DBI32Type*)malloc(sizeof(DBI32Type) * MAT_55c975cf4cf0_size);
DBI32Type* d_MAT_55c975cd41b0aggr__o_shippriority;
hipMalloc(&d_MAT_55c975cd41b0aggr__o_shippriority, sizeof(DBI32Type) * MAT_55c975cf4cf0_size);
main_pipeline_55c975d4fa00<<<std::ceil((float)MAT_55c975cf4cf0_size/(float)32), 32>>>(d_MAT_55c975cd41b0aggr0__tmp_attr0, d_MAT_55c975cd41b0aggr__o_orderdate, d_MAT_55c975cd41b0aggr__o_shippriority, d_MAT_55c975cd41b0lineitem__l_orderkey, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_KEY_55c975cf4cf0lineitem__l_orderkey, MAT_55c975cf4cf0_size);
hipMemcpy(MAT_55c975cd41b0lineitem__l_orderkey, d_MAT_55c975cd41b0lineitem__l_orderkey, sizeof(DBI32Type) * MAT_55c975cf4cf0_size, hipMemcpyDeviceToHost);
hipMemcpy(MAT_55c975cd41b0aggr0__tmp_attr0, d_MAT_55c975cd41b0aggr0__tmp_attr0, sizeof(DBDecimalType) * MAT_55c975cf4cf0_size, hipMemcpyDeviceToHost);
hipMemcpy(MAT_55c975cd41b0aggr__o_orderdate, d_MAT_55c975cd41b0aggr__o_orderdate, sizeof(DBDateType) * MAT_55c975cf4cf0_size, hipMemcpyDeviceToHost);
hipMemcpy(MAT_55c975cd41b0aggr__o_shippriority, d_MAT_55c975cd41b0aggr__o_shippriority, sizeof(DBI32Type) * MAT_55c975cf4cf0_size, hipMemcpyDeviceToHost);
for (auto i=0ull; i< MAT_55c975cf4cf0_size; i++) {
std::cout << MAT_55c975cd41b0lineitem__l_orderkey[i] << "\t";
std::cout << MAT_55c975cd41b0aggr0__tmp_attr0[i] << "\t";
std::cout << MAT_55c975cd41b0aggr__o_orderdate[i] << "\t";
std::cout << MAT_55c975cd41b0aggr__o_shippriority[i] << "\t";
std::cout << std::endl;}

}
