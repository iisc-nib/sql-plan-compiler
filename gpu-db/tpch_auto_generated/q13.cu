#include "hip/hip_runtime.h"
#include "utils.h"

#include <cuco/static_map.cuh>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <chrono>

namespace cg = cooperative_groups;

__global__ void pipeline_1(int32_t *c_custkey, int32_t *o_custkey, int64_t *B0_idx, size_t customer_size)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= customer_size)
        return;
    int32_t reg_c_custkey = c_custkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_c_custkey) << 0);
    atomicAdd((int *)B0_idx, 1);
}

template <typename TY_HT0_I, typename TY_HT0_F>
__global__ void pipeline_0(int32_t *c_custkey, int32_t *o_custkey, TY_HT0_I HT0_I, TY_HT0_F HT0_F, int64_t *B0_idx, size_t customer_size, int64_t *B0_customer)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= customer_size)
        return;
    int32_t reg_c_custkey = c_custkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_c_custkey) << 0);
    auto reg_B0_idx = atomicAdd((int *)B0_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT0_I.insert(thread, cuco::pair{key0, reg_B0_idx});
    B0_customer[reg_B0_idx] = tid;
}

template <typename TY_HT1_I, typename TY_HT1_F>
__global__ void pipeline_2(TY_HT1_I HT1_I, TY_HT1_F HT1_F, int32_t *c_custkey, size_t customer_size)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= customer_size)
        return;
    int32_t reg_c_custkey = c_custkey[tid];
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_c_custkey) << 0);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT1_I.insert(thread, cuco::pair{key1, 1});
}

template <typename TY_HT0_I, typename TY_HT0_F, typename TY_HT1_I, typename TY_HT1_F>
__global__ void pipeline_3(char *o_comment, int64_t *o_comment_offsets, int *o_comment_sizes, int32_t *c_custkey, int32_t *o_custkey, int32_t *agg_custkey, int64_t *agg_count, TY_HT0_I HT0_I, TY_HT0_F HT0_F, TY_HT1_I HT1_I, TY_HT1_F HT1_F, size_t orders_size, int64_t *B0_customer)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= orders_size)
        return;
    if ((like_operator(o_comment + o_comment_offsets[tid], o_comment_sizes[tid], "special", 7)))
        return;
    int32_t reg_o_custkey = o_custkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_o_custkey) << 0);
    auto slot0 = HT0_F.find(key0);
    if (slot0 == HT0_F.end())
        return;
    int32_t reg_c_custkey = c_custkey[B0_customer[slot0->second]];
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_c_custkey) << 0);
    auto slot1 = HT1_F.find(key1);
    agg_custkey[slot1->second] = reg_c_custkey;
    aggregate_sum(&(agg_count[slot1->second]), 1);
}

template <typename TY_HT2_I, typename TY_HT2_F>
__global__ void pipeline_4(int64_t *agg_count, TY_HT2_I HT2_I, TY_HT2_F HT2_F, size_t agg_size)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= agg_size)
        return;
    int64_t reg_agg_count = agg_count[tid];
    int64_t key2 = 0;
    key2 |= (((int64_t)reg_agg_count) << 0);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT2_I.insert(thread, cuco::pair{key2, 1});
}

template <typename TY_HT2_I, typename TY_HT2_F>
__global__ void pipeline_5(int64_t *agg_count, int64_t *agg2_count, int64_t *agg2_dist, TY_HT2_I HT2_I, TY_HT2_F HT2_F, size_t agg_size)
{
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= agg_size)
        return;
    int64_t reg_agg_count = agg_count[tid];
    int64_t key2 = 0;
    key2 |= (((int64_t)reg_agg_count) << 0);
    auto slot2 = HT2_F.find(key2);
    agg2_count[slot2->second] = reg_agg_count;
    aggregate_sum(&(agg2_dist[slot2->second]), 1);
}

void control(
    int32_t *c_custkey,
    StringColumn *o_comment,
    int32_t *o_custkey,
    size_t customer_size,
    size_t orders_size)
{
    int32_t *d_c_custkey;
    auto start = std::chrono::high_resolution_clock::now();

    hipMalloc(&d_c_custkey, sizeof(int32_t) * customer_size);
    
    hipMemcpy(d_c_custkey, c_custkey, sizeof(int32_t) * customer_size, hipMemcpyHostToDevice);
    
    int32_t *d_o_custkey;
    
    hipMalloc(&d_o_custkey, sizeof(int32_t) * orders_size);
    
    hipMemcpy(d_o_custkey, o_custkey, sizeof(int32_t) * orders_size, hipMemcpyHostToDevice);
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
    std::cout << "mallocs time: " << duration.count() << " ms.\n";

    int64_t *B0_customer;
    int64_t *B0_idx;
    hipMalloc(&B0_idx, sizeof(int64_t));
    hipMemset(B0_idx, 0, sizeof(int64_t));
    pipeline_1<<<std::ceil((float)customer_size / (float)32), 32>>>(d_c_custkey, d_o_custkey, B0_idx, customer_size);

    int64_t h_B0_idx;
    hipMemcpy(&h_B0_idx, B0_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B0_idx, 0, sizeof(int64_t));
    hipMalloc(&B0_customer, sizeof(int64_t) * h_B0_idx);
    auto HT0 = cuco::static_map{h_B0_idx * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT0_F = HT0.ref(cuco::find);

    auto d_HT0_I = HT0.ref(cuco::insert);

    pipeline_0<<<std::ceil((float)customer_size / (float)32), 32>>>(d_c_custkey, d_o_custkey, d_HT0_I, d_HT0_F, B0_idx, customer_size, B0_customer);

    char *d_o_comment;

    int64_t *d_o_comment_offsets;

    int *d_o_comment_sizes;

    hipMalloc(&d_o_comment, sizeof(char) * (o_comment->offsets[orders_size - 1] + o_comment->sizes[orders_size - 1]));

    hipMemcpy(d_o_comment, o_comment->data, sizeof(char) * (o_comment->offsets[orders_size - 1] + o_comment->sizes[orders_size - 1]), hipMemcpyHostToDevice);

    hipMalloc(&d_o_comment_offsets, sizeof(int64_t) * orders_size);

    hipMemcpy(d_o_comment_offsets, o_comment->offsets, sizeof(int64_t) * orders_size, hipMemcpyHostToDevice);

    hipMalloc(&d_o_comment_sizes, sizeof(int) * orders_size);

    hipMemcpy(d_o_comment_sizes, o_comment->sizes, sizeof(int) * orders_size, hipMemcpyHostToDevice);

    int32_t *d_agg_custkey;

    int64_t *d_agg_count;

    auto HT1 = cuco::static_map{orders_size * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT1_F = HT1.ref(cuco::find);

    auto d_HT1_I = HT1.ref(cuco::insert);

    pipeline_2<<<std::ceil((float)orders_size / (float)32), 32>>>(d_HT1_I, d_HT1_F, d_c_custkey, customer_size);

    auto HT1_size = HT1.size();

    hipMalloc(&d_agg_custkey, sizeof(int32_t) * HT1_size);

    hipMalloc(&d_agg_count, sizeof(int64_t) * HT1_size);

    hipMemset(d_agg_custkey, 0, sizeof(int32_t) * HT1_size);

    hipMemset(d_agg_count, 0, sizeof(int64_t) * HT1_size);

    thrust::device_vector<int64_t> keys_1(HT1_size), vals_1(HT1_size);
    HT1.retrieve_all(keys_1.begin(), vals_1.begin());
    thrust::host_vector<int64_t> h_keys_1(HT1_size);
    thrust::copy(keys_1.begin(), keys_1.end(), h_keys_1.begin());
    thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_1(HT1_size);
    for (int i = 0; i < HT1_size; i++)
    {
        actual_dict_1[i] = cuco::make_pair(h_keys_1[i], i);
    }
    HT1.clear();
    HT1.insert(actual_dict_1.begin(), actual_dict_1.end());
    pipeline_3<<<std::ceil((float)orders_size / (float)32), 32>>>(d_o_comment, d_o_comment_offsets, d_o_comment_sizes, d_c_custkey, d_o_custkey, d_agg_custkey, d_agg_count, d_HT0_I, d_HT0_F, d_HT1_I, d_HT1_F, orders_size, B0_customer);

    size_t agg_size = HT1_size;
    
    int64_t *d_agg2_count;

    int64_t *d_agg2_dist;

    auto HT2 = cuco::static_map{agg_size * 2, cuco::empty_key{(int64_t)-1}, cuco::empty_value{(int64_t)-1}, thrust::equal_to<int64_t>{}, cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

    auto d_HT2_F = HT2.ref(cuco::find);

    auto d_HT2_I = HT2.ref(cuco::insert);

    pipeline_4<<<std::ceil((float)agg_size / (float)32), 32>>>(d_agg_count, d_HT2_I, d_HT2_F, agg_size);

    auto HT2_size = HT2.size();

    hipMalloc(&d_agg2_count, sizeof(int64_t) * HT2_size);

    hipMalloc(&d_agg2_dist, sizeof(int64_t) * HT2_size);

    hipMemset(d_agg2_count, 0, sizeof(int64_t) * HT2_size);

    hipMemset(d_agg2_dist, 0, sizeof(int64_t) * HT2_size);

    thrust::device_vector<int64_t> keys_2(HT2_size), vals_2(HT2_size);
    HT2.retrieve_all(keys_2.begin(), vals_2.begin());
    thrust::host_vector<int64_t> h_keys_2(HT2_size);
    thrust::copy(keys_2.begin(), keys_2.end(), h_keys_2.begin());
    thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_2(HT2_size);
    for (int i = 0; i < HT2_size; i++)
    {
        actual_dict_2[i] = cuco::make_pair(h_keys_2[i], i);
    }
    HT2.clear();
    HT2.insert(actual_dict_2.begin(), actual_dict_2.end());
    pipeline_5<<<std::ceil((float)agg_size / (float)32), 32>>>(d_agg_count, d_agg2_count, d_agg2_dist, d_HT2_I, d_HT2_F, agg_size);
    size_t agg2_size = HT2_size;
    int64_t *p_agg2_count = (int64_t *)malloc(sizeof(int64_t) * agg2_size);
    hipMemcpy(p_agg2_count, d_agg2_count, sizeof(int64_t) * agg2_size, hipMemcpyDeviceToHost);
    int64_t *p_agg2_dist = (int64_t *)malloc(sizeof(int64_t) * agg2_size);
    hipMemcpy(p_agg2_dist, d_agg2_dist, sizeof(int64_t) * agg2_size, hipMemcpyDeviceToHost);
    for (int i = 0; i < agg2_size; i++)
    {
        std::cout << p_agg2_count[i] << "\t";
        std::cout << p_agg2_dist[i] << "\t";
        std::cout << std::endl;
    }
}
int main(int argc, const char **argv)
{
    std::string dbDir = getDataDir(argv, argc);
    std::string customer_file = dbDir + "customer.parquet";
    auto customer_table = getArrowTable(customer_file);
    size_t customer_size = customer_table->num_rows();
    std::string orders_file = dbDir + "orders.parquet";
    auto orders_table = getArrowTable(orders_file);
    size_t orders_size = orders_table->num_rows();
    auto c_custkey = read_column_typecasted<int32_t>(customer_table, "c_custkey");
    auto o_comment = read_string_column(orders_table, "o_comment");
    auto o_custkey = read_column_typecasted<int32_t>(orders_table, "o_custkey");
    auto start = std::chrono::high_resolution_clock::now();
    control(c_custkey.data(), o_comment, o_custkey.data(), customer_size, orders_size);
    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
    std::cout << "control execution time: " << duration.count() << " ms.\n";
}
