#include "hip/hip_runtime.h"
#include "utils.h"

#include <cuco/static_map.cuh>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
namespace cg = cooperative_groups;

__global__ void pipeline_1 (int32_t* c_nationkey, int32_t* l_suppkey, int32_t* s_suppkey, int32_t* s_nationkey, size_t supplier_size, int64_t* B4_idx) {
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= supplier_size) return;
    int32_t reg_s_nationkey = s_nationkey[tid];
    int32_t reg_s_suppkey = s_suppkey[tid];
    int64_t key4 = 0;
    key4 |= (((int64_t)reg_s_nationkey) << 0);
    key4 |= (((int64_t)reg_s_suppkey) << 32);
    atomicAdd((int*)B4_idx, 1);
    }
    
    template <typename TY_HT4_I, typename TY_HT4_F>
    __global__ void pipeline_0 (int32_t* c_nationkey, int32_t* l_suppkey, int32_t* s_suppkey, int32_t* s_nationkey, TY_HT4_I HT4_I, TY_HT4_F HT4_F, size_t supplier_size, int64_t* B4_supplier, int64_t* B4_idx) {
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= supplier_size) return;
    int32_t reg_s_nationkey = s_nationkey[tid];
    int32_t reg_s_suppkey = s_suppkey[tid];
    int64_t key4 = 0;
    key4 |= (((int64_t)reg_s_nationkey) << 0);
    key4 |= (((int64_t)reg_s_suppkey) << 32);
    auto reg_B4_idx = atomicAdd((int*)B4_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT4_I.insert(thread, cuco::pair{key4, reg_B4_idx});
    B4_supplier[reg_B4_idx] = tid;
    }
    
    
    __global__ void pipeline_3 (int32_t* r_regionkey, int32_t* n_regionkey, int8_t* r_name, size_t region_size, int64_t* B0_idx) {
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= region_size) return;
    int8_t reg_r_name = r_name[tid];
    if (!(reg_r_name  == 2)) return;
    int32_t reg_r_regionkey = r_regionkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_r_regionkey) << 0);
    atomicAdd((int*)B0_idx, 1);
    }
    
    template <typename TY_HT0_I, typename TY_HT0_F>
    __global__ void pipeline_2 (int32_t* r_regionkey, int32_t* n_regionkey, int8_t* r_name, TY_HT0_I HT0_I, TY_HT0_F HT0_F, int64_t* B0_region, size_t region_size, int64_t* B0_idx) {
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= region_size) return;
    int8_t reg_r_name = r_name[tid];
    if (!(reg_r_name  == 2)) return;
    int32_t reg_r_regionkey = r_regionkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_r_regionkey) << 0);
    auto reg_B0_idx = atomicAdd((int*)B0_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT0_I.insert(thread, cuco::pair{key0, reg_B0_idx});
    B0_region[reg_B0_idx] = tid;
    }
    
    template <typename TY_HT0_I, typename TY_HT0_F>
    __global__ void pipeline_5 (int32_t* c_nationkey, int32_t* n_regionkey, int32_t* n_nationkey, TY_HT0_I HT0_I, TY_HT0_F HT0_F, int64_t* B0_region, size_t nation_size, int64_t* B1_idx) {
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= nation_size) return;
    int32_t reg_n_regionkey = n_regionkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_n_regionkey) << 0);
    auto slot0 = HT0_F.find(key0);
    if (slot0 == HT0_F.end()) return;
    int32_t reg_n_nationkey = n_nationkey[tid];
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_n_nationkey) << 0);
    atomicAdd((int*)B1_idx, 1);
    }
    
    template <typename TY_HT0_I, typename TY_HT0_F, typename TY_HT1_I, typename TY_HT1_F>
    __global__ void pipeline_4 (int32_t* c_nationkey, int32_t* n_regionkey, int32_t* n_nationkey, TY_HT0_I HT0_I, TY_HT0_F HT0_F, TY_HT1_I HT1_I, TY_HT1_F HT1_F, int64_t* B0_region, int64_t* B1_region, int64_t* B1_nation, size_t nation_size, int64_t* B1_idx) {
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= nation_size) return;
    int32_t reg_n_regionkey = n_regionkey[tid];
    int64_t key0 = 0;
    key0 |= (((int64_t)reg_n_regionkey) << 0);
    auto slot0 = HT0_F.find(key0);
    if (slot0 == HT0_F.end()) return;
    int32_t reg_n_nationkey = n_nationkey[tid];
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_n_nationkey) << 0);
    auto reg_B1_idx = atomicAdd((int*)B1_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT1_I.insert(thread, cuco::pair{key1, reg_B1_idx});
    B1_region[reg_B1_idx] = B0_region[slot0->second];
    B1_nation[reg_B1_idx] = tid;
    }
    
    template <typename TY_HT1_I, typename TY_HT1_F>
    __global__ void pipeline_7 (int32_t* c_custkey, int32_t* c_nationkey, int32_t* o_custkey, TY_HT1_I HT1_I, TY_HT1_F HT1_F, size_t customer_size, int64_t* B1_region, int64_t* B1_nation, int64_t* B2_idx) {
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= customer_size) return;
    int32_t reg_c_nationkey = c_nationkey[tid];
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_c_nationkey) << 0);
    auto slot1 = HT1_F.find(key1);
    if (slot1 == HT1_F.end()) return;
    int32_t reg_c_custkey = c_custkey[tid];
    int64_t key2 = 0;
    key2 |= (((int64_t)reg_c_custkey) << 0);
    atomicAdd((int*)B2_idx, 1);
    }
    
    template <typename TY_HT1_I, typename TY_HT1_F, typename TY_HT2_I, typename TY_HT2_F>
    __global__ void pipeline_6 (int32_t* c_custkey, int32_t* c_nationkey, int32_t* o_custkey, TY_HT1_I HT1_I, TY_HT1_F HT1_F, TY_HT2_I HT2_I, TY_HT2_F HT2_F, int64_t* B1_region, int64_t* B2_nation, size_t customer_size, int64_t* B1_nation, int64_t* B2_region, int64_t* B2_customer, int64_t* B2_idx) {
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= customer_size) return;
    int32_t reg_c_nationkey = c_nationkey[tid];
    int64_t key1 = 0;
    key1 |= (((int64_t)reg_c_nationkey) << 0);
    auto slot1 = HT1_F.find(key1);
    if (slot1 == HT1_F.end()) return;
    int32_t reg_c_custkey = c_custkey[tid];
    int64_t key2 = 0;
    key2 |= (((int64_t)reg_c_custkey) << 0);
    auto reg_B2_idx = atomicAdd((int*)B2_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT2_I.insert(thread, cuco::pair{key2, reg_B2_idx});
    B2_region[reg_B2_idx] = B1_region[slot1->second];
    B2_customer[reg_B2_idx] = tid;
    B2_nation[reg_B2_idx] = B1_nation[slot1->second];
    }
    
    template <typename TY_HT2_I, typename TY_HT2_F>
    __global__ void pipeline_9 (int32_t* o_orderdate, int32_t* o_custkey, int32_t* l_orderkey, int32_t* o_orderkey, TY_HT2_I HT2_I, TY_HT2_F HT2_F, int64_t* B3_idx, int64_t* B2_nation, int64_t* B2_customer, size_t orders_size, int64_t* B2_region) {
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= orders_size) return;
    int32_t reg_o_orderdate = o_orderdate[tid];
    if (!(reg_o_orderdate >= 8766)) return;
    if (!(reg_o_orderdate < 9131)) return;
    int32_t reg_o_custkey = o_custkey[tid];
    int64_t key2 = 0;
    key2 |= (((int64_t)reg_o_custkey) << 0);
    auto slot2 = HT2_F.find(key2);
    if (slot2 == HT2_F.end()) return;
    int32_t reg_o_orderkey = o_orderkey[tid];
    int64_t key3 = 0;
    key3 |= (((int64_t)reg_o_orderkey) << 0);
    atomicAdd((int*)B3_idx, 1);
    }
    
    template <typename TY_HT2_I, typename TY_HT2_F, typename TY_HT3_I, typename TY_HT3_F>
    __global__ void pipeline_8 (int32_t* o_orderdate, int32_t* o_custkey, int32_t* l_orderkey, int32_t* o_orderkey, TY_HT2_I HT2_I, TY_HT2_F HT2_F, TY_HT3_I HT3_I, TY_HT3_F HT3_F, int64_t* B3_idx, int64_t* B2_nation, int64_t* B2_customer, int64_t* B2_region, size_t orders_size, int64_t* B3_orders, int64_t* B3_nation, int64_t* B3_customer, int64_t* B3_region) {
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= orders_size) return;
    int32_t reg_o_orderdate = o_orderdate[tid];
    if (!(reg_o_orderdate >= 8766)) return;
    if (!(reg_o_orderdate < 9131)) return;
    int32_t reg_o_custkey = o_custkey[tid];
    int64_t key2 = 0;
    key2 |= (((int64_t)reg_o_custkey) << 0);
    auto slot2 = HT2_F.find(key2);
    if (slot2 == HT2_F.end()) return;
    int32_t reg_o_orderkey = o_orderkey[tid];
    int64_t key3 = 0;
    key3 |= (((int64_t)reg_o_orderkey) << 0);
    auto reg_B3_idx = atomicAdd((int*)B3_idx, 1);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT3_I.insert(thread, cuco::pair{key3, reg_B3_idx});
    B3_region[reg_B3_idx] = B2_region[slot2->second];
    B3_nation[reg_B3_idx] = B2_nation[slot2->second];
    B3_customer[reg_B3_idx] = B2_customer[slot2->second];
    B3_orders[reg_B3_idx] = tid;
    }
    
    template <typename TY_HT3_I, typename TY_HT3_F, typename TY_HT4_I, typename TY_HT4_F, typename TY_HT5_I, typename TY_HT5_F>
    __global__ void pipeline_10 (int32_t* l_suppkey, int32_t* l_orderkey, double* l_discount, int8_t* n_name, int32_t* c_nationkey, double* l_extendedprice, TY_HT3_I HT3_I, TY_HT3_F HT3_F, TY_HT4_I HT4_I, TY_HT4_F HT4_F, TY_HT5_I HT5_I, TY_HT5_F HT5_F, size_t lineitem_size, int64_t* B4_supplier, int64_t* B3_orders, int64_t* B3_nation, int64_t* B3_customer, int64_t* B3_region) {
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= lineitem_size) return;
    int32_t reg_l_orderkey = l_orderkey[tid];
    int64_t key3 = 0;
    key3 |= (((int64_t)reg_l_orderkey) << 0);
    auto slot3 = HT3_F.find(key3);
    if (slot3 == HT3_F.end()) return;
    int32_t reg_c_nationkey = c_nationkey[B3_customer[slot3->second]];
    int32_t reg_l_suppkey = l_suppkey[tid];
    int64_t key4 = 0;
    key4 |= (((int64_t)reg_c_nationkey) << 0);
    key4 |= (((int64_t)reg_l_suppkey) << 32);
    auto slot4 = HT4_F.find(key4);
    if (slot4 == HT4_F.end()) return;
    double reg_l_extendedprice = l_extendedprice[tid];
    double reg_l_discount = l_discount[tid];
    double reg_revenue = (reg_l_extendedprice * (1 - reg_l_discount));
    int8_t reg_n_name = n_name[B3_nation[slot3->second]];
    int64_t key5 = 0;
    key5 |= (((int64_t)reg_n_name) << 0);
    auto thread = cg::tiled_partition<1>(cg::this_thread_block());
    HT5_I.insert(thread, cuco::pair{key5, 1});
    }
    
    template <typename TY_HT3_I, typename TY_HT3_F, typename TY_HT4_I, typename TY_HT4_F, typename TY_HT5_I, typename TY_HT5_F>
    __global__ void pipeline_11 (int32_t* l_suppkey, double* l_extendedprice, int32_t* l_orderkey, double* l_discount, int32_t* c_nationkey, int8_t* n_name, int8_t* agg_n_name, double* agg_revenue, TY_HT3_I HT3_I, TY_HT3_F HT3_F, TY_HT4_I HT4_I, TY_HT4_F HT4_F, TY_HT5_I HT5_I, TY_HT5_F HT5_F, int64_t* B4_supplier, size_t lineitem_size, int64_t* B3_orders, int64_t* B3_nation, int64_t* B3_customer, int64_t* B3_region) {
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= lineitem_size) return;
    int32_t reg_l_orderkey = l_orderkey[tid];
    int64_t key3 = 0;
    key3 |= (((int64_t)reg_l_orderkey) << 0);
    auto slot3 = HT3_F.find(key3);
    if (slot3 == HT3_F.end()) return;
    int32_t reg_c_nationkey = c_nationkey[B3_customer[slot3->second]];
    int32_t reg_l_suppkey = l_suppkey[tid];
    int64_t key4 = 0;
    key4 |= (((int64_t)reg_c_nationkey) << 0);
    key4 |= (((int64_t)reg_l_suppkey) << 32);
    auto slot4 = HT4_F.find(key4);
    if (slot4 == HT4_F.end()) return;
    double reg_l_extendedprice = l_extendedprice[tid];
    double reg_l_discount = l_discount[tid];
    double reg_revenue = (reg_l_extendedprice * (1 - reg_l_discount));
    int8_t reg_n_name = n_name[B3_nation[slot3->second]];
    int64_t key5 = 0;
    key5 |= (((int64_t)reg_n_name) << 0);
    auto slot5 = HT5_F.find(key5);
    agg_n_name[slot5->second] = reg_n_name;
    aggregate_sum(&(agg_revenue[slot5->second]), reg_revenue);
    }
    
    void control(
    int32_t *c_custkey,
    int32_t *c_nationkey,
    double *l_discount,
    double *l_extendedprice,
    int32_t *l_orderkey,
    int32_t *l_suppkey,
    int8_t *n_name,
    int32_t *n_nationkey,
    int32_t *n_regionkey,
    int32_t *o_custkey,
    int32_t *o_orderdate,
    int32_t *o_orderkey,
    int8_t *r_name,
    int32_t *r_regionkey,
    int32_t *s_nationkey,
    int32_t *s_suppkey,
    size_t customer_size,
    size_t lineitem_size,
    size_t nation_size,
    size_t orders_size,
    size_t region_size,
    size_t supplier_size
    ) {
    int32_t* d_c_nationkey;
    
    hipMalloc(&d_c_nationkey, sizeof(int32_t) * customer_size);
    
    hipMemcpy(d_c_nationkey, c_nationkey, sizeof(int32_t) * customer_size, hipMemcpyHostToDevice);
    
    int32_t* d_l_suppkey;
    
    hipMalloc(&d_l_suppkey, sizeof(int32_t) * lineitem_size);
    
    hipMemcpy(d_l_suppkey, l_suppkey, sizeof(int32_t) * lineitem_size, hipMemcpyHostToDevice);
    
    int32_t* d_s_suppkey;
    
    hipMalloc(&d_s_suppkey, sizeof(int32_t) * supplier_size);
    
    hipMemcpy(d_s_suppkey, s_suppkey, sizeof(int32_t) * supplier_size, hipMemcpyHostToDevice);
    
    int32_t* d_s_nationkey;
    
    hipMalloc(&d_s_nationkey, sizeof(int32_t) * supplier_size);
    
    hipMemcpy(d_s_nationkey, s_nationkey, sizeof(int32_t) * supplier_size, hipMemcpyHostToDevice);
    
    int64_t* B4_supplier;
    int64_t* B4_idx;
    hipMalloc(&B4_idx, sizeof(int64_t));
    hipMemset(B4_idx, 0, sizeof(int64_t));
    pipeline_1<<<std::ceil((float)supplier_size/(float)32), 32>>>(d_c_nationkey, d_l_suppkey, d_s_suppkey, d_s_nationkey, supplier_size, B4_idx);
    
    int64_t h_B4_idx;
    hipMemcpy(&h_B4_idx, B4_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B4_idx, 0, sizeof(int64_t));
    hipMalloc(&B4_supplier, sizeof(int64_t) * h_B4_idx);
    auto HT4 = cuco::static_map{ h_B4_idx * 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};
    
    auto d_HT4_F = HT4.ref(cuco::find);
    
    auto d_HT4_I = HT4.ref(cuco::insert);
    
    pipeline_0<<<std::ceil((float)supplier_size/(float)32), 32>>>(d_c_nationkey, d_l_suppkey, d_s_suppkey, d_s_nationkey, d_HT4_I, d_HT4_F, supplier_size, B4_supplier, B4_idx);
    
    int32_t* d_r_regionkey;
    
    hipMalloc(&d_r_regionkey, sizeof(int32_t) * region_size);
    
    hipMemcpy(d_r_regionkey, r_regionkey, sizeof(int32_t) * region_size, hipMemcpyHostToDevice);
    
    int32_t* d_n_regionkey;
    
    hipMalloc(&d_n_regionkey, sizeof(int32_t) * nation_size);
    
    hipMemcpy(d_n_regionkey, n_regionkey, sizeof(int32_t) * nation_size, hipMemcpyHostToDevice);
    
    int8_t* d_r_name;
    
    hipMalloc(&d_r_name, sizeof(int8_t) * region_size);
    
    hipMemcpy(d_r_name, r_name, sizeof(int8_t) * region_size, hipMemcpyHostToDevice);
    
    int64_t* B0_region;
    int64_t* B0_idx;
    hipMalloc(&B0_idx, sizeof(int64_t));
    hipMemset(B0_idx, 0, sizeof(int64_t));
    pipeline_3<<<std::ceil((float)region_size/(float)32), 32>>>(d_r_regionkey, d_n_regionkey, d_r_name, region_size, B0_idx);
    
    int64_t h_B0_idx;
    hipMemcpy(&h_B0_idx, B0_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B0_idx, 0, sizeof(int64_t));
    hipMalloc(&B0_region, sizeof(int64_t) * h_B0_idx);
    auto HT0 = cuco::static_map{ h_B0_idx * 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};
    
    auto d_HT0_F = HT0.ref(cuco::find);
    
    auto d_HT0_I = HT0.ref(cuco::insert);
    
    pipeline_2<<<std::ceil((float)region_size/(float)32), 32>>>(d_r_regionkey, d_n_regionkey, d_r_name, d_HT0_I, d_HT0_F, B0_region, region_size, B0_idx);
    
    int32_t* d_n_nationkey;
    
    hipMalloc(&d_n_nationkey, sizeof(int32_t) * nation_size);
    
    hipMemcpy(d_n_nationkey, n_nationkey, sizeof(int32_t) * nation_size, hipMemcpyHostToDevice);
    
    int64_t* B1_region;
    int64_t* B1_nation;
    int64_t* B1_idx;
    hipMalloc(&B1_idx, sizeof(int64_t));
    hipMemset(B1_idx, 0, sizeof(int64_t));
    pipeline_5<<<std::ceil((float)nation_size/(float)32), 32>>>(d_c_nationkey, d_n_regionkey, d_n_nationkey, d_HT0_I, d_HT0_F, B0_region, nation_size, B1_idx);
    
    int64_t h_B1_idx;
    hipMemcpy(&h_B1_idx, B1_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B1_idx, 0, sizeof(int64_t));
    hipMalloc(&B1_region, sizeof(int64_t) * h_B1_idx);
    hipMalloc(&B1_nation, sizeof(int64_t) * h_B1_idx);
    auto HT1 = cuco::static_map{ h_B1_idx * 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};
    
    auto d_HT1_F = HT1.ref(cuco::find);
    
    auto d_HT1_I = HT1.ref(cuco::insert);
    
    pipeline_4<<<std::ceil((float)nation_size/(float)32), 32>>>(d_c_nationkey, d_n_regionkey, d_n_nationkey, d_HT0_I, d_HT0_F, d_HT1_I, d_HT1_F, B0_region, B1_region, B1_nation, nation_size, B1_idx);
    
    int32_t* d_c_custkey;
    
    hipMalloc(&d_c_custkey, sizeof(int32_t) * customer_size);
    
    hipMemcpy(d_c_custkey, c_custkey, sizeof(int32_t) * customer_size, hipMemcpyHostToDevice);
    
    int32_t* d_o_custkey;
    
    hipMalloc(&d_o_custkey, sizeof(int32_t) * orders_size);
    
    hipMemcpy(d_o_custkey, o_custkey, sizeof(int32_t) * orders_size, hipMemcpyHostToDevice);
    
    int64_t* B2_region;
    int64_t* B2_customer;
    int64_t* B2_nation;
    int64_t* B2_idx;
    hipMalloc(&B2_idx, sizeof(int64_t));
    hipMemset(B2_idx, 0, sizeof(int64_t));
    pipeline_7<<<std::ceil((float)customer_size/(float)32), 32>>>(d_c_custkey, d_c_nationkey, d_o_custkey, d_HT1_I, d_HT1_F, customer_size, B1_region, B1_nation, B2_idx);
    
    int64_t h_B2_idx;
    hipMemcpy(&h_B2_idx, B2_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B2_idx, 0, sizeof(int64_t));
    hipMalloc(&B2_region, sizeof(int64_t) * h_B2_idx);
    hipMalloc(&B2_customer, sizeof(int64_t) * h_B2_idx);
    hipMalloc(&B2_nation, sizeof(int64_t) * h_B2_idx);
    auto HT2 = cuco::static_map{ h_B2_idx * 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};
    
    auto d_HT2_F = HT2.ref(cuco::find);
    
    auto d_HT2_I = HT2.ref(cuco::insert);
    
    pipeline_6<<<std::ceil((float)customer_size/(float)32), 32>>>(d_c_custkey, d_c_nationkey, d_o_custkey, d_HT1_I, d_HT1_F, d_HT2_I, d_HT2_F, B1_region, B2_nation, customer_size, B1_nation, B2_region, B2_customer, B2_idx);
    
    int32_t* d_o_orderdate;
    
    hipMalloc(&d_o_orderdate, sizeof(int32_t) * orders_size);
    
    hipMemcpy(d_o_orderdate, o_orderdate, sizeof(int32_t) * orders_size, hipMemcpyHostToDevice);
    
    int32_t* d_l_orderkey;
    
    hipMalloc(&d_l_orderkey, sizeof(int32_t) * lineitem_size);
    
    hipMemcpy(d_l_orderkey, l_orderkey, sizeof(int32_t) * lineitem_size, hipMemcpyHostToDevice);
    
    int32_t* d_o_orderkey;
    
    hipMalloc(&d_o_orderkey, sizeof(int32_t) * orders_size);
    
    hipMemcpy(d_o_orderkey, o_orderkey, sizeof(int32_t) * orders_size, hipMemcpyHostToDevice);
    
    int64_t* B3_region;
    int64_t* B3_nation;
    int64_t* B3_customer;
    int64_t* B3_orders;
    int64_t* B3_idx;
    hipMalloc(&B3_idx, sizeof(int64_t));
    hipMemset(B3_idx, 0, sizeof(int64_t));
    pipeline_9<<<std::ceil((float)orders_size/(float)32), 32>>>(d_o_orderdate, d_o_custkey, d_l_orderkey, d_o_orderkey, d_HT2_I, d_HT2_F, B3_idx, B2_nation, B2_customer, orders_size, B2_region);
    
    int64_t h_B3_idx;
    hipMemcpy(&h_B3_idx, B3_idx, sizeof(int64_t), hipMemcpyDeviceToHost);
    hipMemset(B3_idx, 0, sizeof(int64_t));
    hipMalloc(&B3_orders, sizeof(int64_t) * h_B3_idx);
    hipMalloc(&B3_region, sizeof(int64_t) * h_B3_idx);
    hipMalloc(&B3_customer, sizeof(int64_t) * h_B3_idx);
    hipMalloc(&B3_nation, sizeof(int64_t) * h_B3_idx);
    auto HT3 = cuco::static_map{ h_B3_idx * 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};
    
    auto d_HT3_F = HT3.ref(cuco::find);
    
    auto d_HT3_I = HT3.ref(cuco::insert);
    
    pipeline_8<<<std::ceil((float)orders_size/(float)32), 32>>>(d_o_orderdate, d_o_custkey, d_l_orderkey, d_o_orderkey, d_HT2_I, d_HT2_F, d_HT3_I, d_HT3_F, B3_idx, B2_nation, B2_customer, B2_region, orders_size, B3_orders, B3_nation, B3_customer, B3_region);
    
    double* d_l_extendedprice;
    
    hipMalloc(&d_l_extendedprice, sizeof(double) * lineitem_size);
    
    hipMemcpy(d_l_extendedprice, l_extendedprice, sizeof(double) * lineitem_size, hipMemcpyHostToDevice);
    
    double* d_l_discount;
    
    hipMalloc(&d_l_discount, sizeof(double) * lineitem_size);
    
    hipMemcpy(d_l_discount, l_discount, sizeof(double) * lineitem_size, hipMemcpyHostToDevice);
    
    int8_t* d_n_name;
    
    hipMalloc(&d_n_name, sizeof(int8_t) * nation_size);
    
    hipMemcpy(d_n_name, n_name, sizeof(int8_t) * nation_size, hipMemcpyHostToDevice);
    
    int8_t* d_agg_n_name;
    
    double* d_agg_revenue;
    
    auto HT5 = cuco::static_map{ lineitem_size * 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};
    
    auto d_HT5_F = HT5.ref(cuco::find);
    
    auto d_HT5_I = HT5.ref(cuco::insert);
    
    pipeline_10<<<std::ceil((float)lineitem_size/(float)32), 32>>>(d_l_suppkey, d_l_orderkey, d_l_discount, d_n_name, d_c_nationkey, d_l_extendedprice, d_HT3_I, d_HT3_F, d_HT4_I, d_HT4_F, d_HT5_I, d_HT5_F, lineitem_size, B4_supplier, B3_orders, B3_nation, B3_customer, B3_region);
    
    auto HT5_size = HT5.size();
    
    hipMalloc(&d_agg_n_name, sizeof(int8_t) * HT5_size);
    
    hipMalloc(&d_agg_revenue, sizeof(double) * HT5_size);
    
    hipMemset(d_agg_n_name, 0, sizeof(int8_t) * HT5_size);
    
    hipMemset(d_agg_revenue, 0, sizeof(double) * HT5_size);
    
    thrust::device_vector<int64_t> keys_5(HT5_size), vals_5(HT5_size);
    HT5.retrieve_all(keys_5.begin(), vals_5.begin());
    thrust::host_vector<int64_t> h_keys_5(HT5_size);
    thrust::copy(keys_5.begin(), keys_5.end(), h_keys_5.begin());
    thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_5(HT5_size);
    for (int i=0; i < HT5_size; i++) {
    actual_dict_5[i] = cuco::make_pair(h_keys_5[i], i);
    }
    HT5.clear();
    HT5.insert(actual_dict_5.begin(), actual_dict_5.end());
    pipeline_11<<<std::ceil((float)lineitem_size/(float)32), 32>>>(d_l_suppkey, d_l_extendedprice, d_l_orderkey, d_l_discount, d_c_nationkey, d_n_name, d_agg_n_name, d_agg_revenue, d_HT3_I, d_HT3_F, d_HT4_I, d_HT4_F, d_HT5_I, d_HT5_F, B4_supplier, lineitem_size, B3_orders, B3_nation, B3_customer, B3_region);
    
    size_t agg_size = HT5_size;
    int8_t* p_agg_n_name = (int8_t*)malloc(sizeof(int8_t) * agg_size);
    hipMemcpy(p_agg_n_name, d_agg_n_name, sizeof(int8_t) * agg_size, hipMemcpyDeviceToHost);
    double* p_agg_revenue = (double*)malloc(sizeof(double) * agg_size);
    hipMemcpy(p_agg_revenue, d_agg_revenue, sizeof(double) * agg_size, hipMemcpyDeviceToHost);
    for (int i=0; i<agg_size; i++) {
    std::cout << (int)p_agg_n_name[i] << "\t";
    std::cout << p_agg_revenue[i] << "\t";
    std::cout << std::endl;
    }
    }
    int main(int argc, const char **argv) {
    std::string dbDir = getDataDir(argv, argc);
    std::string customer_file = dbDir + "customer.parquet";
    auto customer_table = getArrowTable(customer_file);
    size_t customer_size = customer_table->num_rows();
    std::string lineitem_file = dbDir + "lineitem.parquet";
    auto lineitem_table = getArrowTable(lineitem_file);
    size_t lineitem_size = lineitem_table->num_rows();
    std::string nation_file = dbDir + "nation.parquet";
    auto nation_table = getArrowTable(nation_file);
    size_t nation_size = nation_table->num_rows();
    std::string orders_file = dbDir + "orders.parquet";
    auto orders_table = getArrowTable(orders_file);
    size_t orders_size = orders_table->num_rows();
    std::string region_file = dbDir + "region.parquet";
    auto region_table = getArrowTable(region_file);
    size_t region_size = region_table->num_rows();
    std::string supplier_file = dbDir + "supplier.parquet";
    auto supplier_table = getArrowTable(supplier_file);
    size_t supplier_size = supplier_table->num_rows();
    auto c_custkey = read_column_typecasted<int32_t>(customer_table, "c_custkey");
    auto c_nationkey = read_column_typecasted<int32_t>(customer_table, "c_nationkey");
    auto l_discount = read_column<double>(lineitem_table, "l_discount");
    auto l_extendedprice = read_column<double>(lineitem_table, "l_extendedprice");
    auto l_orderkey = read_column_typecasted<int32_t>(lineitem_table, "l_orderkey");
    auto l_suppkey = read_column_typecasted<int32_t>(lineitem_table, "l_suppkey");
    StringDictEncodedColumn *n_name = read_string_dict_encoded_column(nation_table, "n_name");
    auto n_nationkey = read_column_typecasted<int32_t>(nation_table, "n_nationkey");
    auto n_regionkey = read_column_typecasted<int32_t>(nation_table, "n_regionkey");
    auto o_custkey = read_column_typecasted<int32_t>(orders_table, "o_custkey");
    auto o_orderdate = read_column<int32_t>(orders_table, "o_orderdate");
    auto o_orderkey = read_column_typecasted<int32_t>(orders_table, "o_orderkey");
    StringDictEncodedColumn *r_name = read_string_dict_encoded_column(region_table, "r_name");
    auto r_regionkey = read_column_typecasted<int32_t>(region_table, "r_regionkey");
    auto s_nationkey = read_column_typecasted<int32_t>(supplier_table, "s_nationkey");
    auto s_suppkey = read_column_typecasted<int32_t>(supplier_table, "s_suppkey");
    control(c_custkey.data(),c_nationkey.data(),l_discount.data(),l_extendedprice.data(),l_orderkey.data(),l_suppkey.data(),n_name->column,n_nationkey.data(),n_regionkey.data(),o_custkey.data(),o_orderdate.data(),o_orderkey.data(),r_name->column,r_regionkey.data(),s_nationkey.data(),s_suppkey.data(),customer_size,lineitem_size,nation_size,orders_size,region_size,supplier_size);
    }
        