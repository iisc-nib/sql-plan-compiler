#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include "cudautils.cuh"
#include <dirent.h>
#include <dlfcn.h>
#include <iomanip>

#include "dbruntime.h"

__global__ void sample(DBStringType *s1, DBStringType *s2, int *res)
{
    if (evaluatePredicate(s1[0], s2[0], Predicate::eq))
        atomicAdd(res, 1);
}

int main(int argc, const char **argv)
{
    std::string dbDir = getDataDir(argv, argc);
    // std::cout << "Arg len: " << argc << std::endl;
    // assert(argc >= 4 && "Need 3 arguments, --data-dir <path to dir> <shared object query>");
    // const char *soFile = argv[3];
    initTpchDb(dbDir);

    std::cout << std::setprecision(12);

    // TODO(avinash): the runtime should listen for sql queries,
    //  compile it with the lingodb toolchain
    //  look for output.cu
    //  compile it with nvcc
    //  link it using dlopen, and execute the control function.

    // void *lib = dlopen("/media/ajayakar/space/src/sql-plan-compiler/gpu-db/tpch_auto_generated/"+ soFile, RTLD_LAZY);
    while (true)
    {

        std::string libPath;
        std::cout << "> ";
        std::cin >> libPath;
        void *lib = dlopen(libPath.c_str(), RTLD_NOW);
        if (!lib)
        {
            fprintf(stderr, "%s\n", dlerror());
        }
        else
        {
            std::cout << "Opening the shared lib was successful!" << std::endl;

            auto control = reinterpret_cast<void (*)(
                DBI32Type *,
                DBStringType *,
                DBI32Type *,
                DBStringType *,
                size_t,
                DBI32Type *,
                DBI32Type *,
                DBStringType *,
                DBStringType *,
                DBStringType *,
                DBDecimalType *,
                DBStringType *,
                size_t,
                DBI32Type *,
                DBI32Type *,
                DBI32Type *,
                DBDecimalType *,
                DBStringType *,
                size_t,
                DBI32Type *,
                DBStringType *,
                DBStringType *,
                DBStringType *,
                DBStringType *,
                DBI32Type *,
                DBStringType *,
                DBDecimalType *,
                DBStringType *,
                size_t,
                DBI32Type *,
                DBI32Type *,
                DBI32Type *,
                DBI64Type *,
                DBDecimalType *,
                DBDecimalType *,
                DBDecimalType *,
                DBDecimalType *,
                DBCharType *,
                DBCharType *,
                DBI32Type *,
                DBI32Type *,
                DBI32Type *,
                DBStringType *,
                DBStringType *,
                DBStringType *,
                size_t,
                DBI32Type *,
                DBCharType *,
                DBI32Type *,
                DBDecimalType *,
                DBI32Type *,
                DBStringType *,
                DBStringType *,
                DBI32Type *,
                DBStringType *,
                size_t,
                DBI32Type *,
                DBStringType *,
                DBStringType *,
                DBI32Type *,
                DBStringType *,
                DBDecimalType *,
                DBStringType *,
                DBStringType *,
                size_t,
                DBI32Type *,
                DBStringType *,
                DBStringType *,
                size_t)>(dlsym(lib, "control"));
            control(
                d_nation__n_nationkey,
                d_nation__n_name,
                d_nation__n_regionkey,
                d_nation__n_comment,
                nation_size,
                d_supplier__s_suppkey,
                d_supplier__s_nationkey,
                d_supplier__s_name,
                d_supplier__s_address,
                d_supplier__s_phone,
                d_supplier__s_acctbal,
                d_supplier__s_comment,
                supplier_size,
                d_partsupp__ps_suppkey,
                d_partsupp__ps_partkey,
                d_partsupp__ps_availqty,
                d_partsupp__ps_supplycost,
                d_partsupp__ps_comment,
                partsupp_size,
                d_part__p_partkey,
                d_part__p_name,
                d_part__p_mfgr,
                d_part__p_brand,
                d_part__p_type,
                d_part__p_size,
                d_part__p_container,
                d_part__p_retailprice,
                d_part__p_comment,
                part_size,
                d_lineitem__l_orderkey,
                d_lineitem__l_partkey,
                d_lineitem__l_suppkey,
                d_lineitem__l_linenumber,
                d_lineitem__l_quantity,
                d_lineitem__l_extendedprice,
                d_lineitem__l_discount,
                d_lineitem__l_tax,
                d_lineitem__l_returnflag,
                d_lineitem__l_linestatus,
                d_lineitem__l_shipdate,
                d_lineitem__l_commitdate,
                d_lineitem__l_receiptdate,
                d_lineitem__l_shipinstruct,
                d_lineitem__l_shipmode,
                d_lineitem__comments,
                lineitem_size,
                d_orders__o_orderkey,
                d_orders__o_orderstatus,
                d_orders__o_custkey,
                d_orders__o_totalprice,
                d_orders__o_orderdate,
                d_orders__o_orderpriority,
                d_orders__o_clerk,
                d_orders__o_shippriority,
                d_orders__o_comment,
                orders_size,
                d_customer__c_custkey,
                d_customer__c_name,
                d_customer__c_address,
                d_customer__c_nationkey,
                d_customer__c_phone,
                d_customer__c_acctbal,
                d_customer__c_mktsegment,
                d_customer__c_comment,
                customer_size,
                d_region__r_regionkey,
                d_region__r_name,
                d_region__r_comment,
                region_size);
        }
        dlclose(lib);
    }
}
