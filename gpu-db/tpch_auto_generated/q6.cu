#include "hip/hip_runtime.h"
#include "utils.h"

#include <cuco/static_map.cuh>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace cg = cooperative_groups;
template <typename TY_HT0_I, typename TY_HT0_F>
__global__ void pipeline_0(int64_t* l_quantity,
                           int32_t* l_shipdate,
                           double* l_discount,
                           double* l_extendedprice,
                           TY_HT0_I HT0_I,
                           TY_HT0_F HT0_F,
                           size_t lineitem_size)
{
  int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid >= lineitem_size) return;
  int32_t reg_l_shipdate = l_shipdate[tid];
  if (!(reg_l_shipdate >= 8766)) return;
  if (!(reg_l_shipdate < 9131)) return;
  int64_t reg_l_quantity = l_quantity[tid];
  if (!(reg_l_quantity < 24)) return;
  double reg_l_discount = l_discount[tid];
  if (!(reg_l_discount >= 0.05)) return;
  if (!(reg_l_discount <= 0.07)) return;
  double reg_l_extendedprice = l_extendedprice[tid];
  double reg_revenue         = (reg_l_extendedprice * reg_l_discount);
  int64_t key0               = 0;
  auto thread                = cg::tiled_partition<1>(cg::this_thread_block());
  HT0_I.insert(thread, cuco::pair{key0, 1});
}

template <typename TY_HT0_I, typename TY_HT0_F>
__global__ void pipeline_1(double* l_extendedprice,
                           int32_t* l_shipdate,
                           int64_t* l_quantity,
                           double* l_discount,
                           double* agg_revenue,
                           TY_HT0_I HT0_I,
                           TY_HT0_F HT0_F,
                           size_t lineitem_size)
{
  int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid >= lineitem_size) return;
  int32_t reg_l_shipdate = l_shipdate[tid];
  if (!(reg_l_shipdate >= 8766)) return;
  if (!(reg_l_shipdate < 9131)) return;
  int64_t reg_l_quantity = l_quantity[tid];
  if (!(reg_l_quantity < 24)) return;
  double reg_l_discount = l_discount[tid];
  if (!(reg_l_discount >= 0.05)) return;
  if (!(reg_l_discount <= 0.07)) return;
  double reg_l_extendedprice = l_extendedprice[tid];
  double reg_revenue         = (reg_l_extendedprice * reg_l_discount);
  int64_t key0               = 0;
  auto slot0                 = HT0_F.find(key0);
  aggregate_sum(&(agg_revenue[slot0->second]), reg_revenue);
}

void control(double* l_discount,
             double* l_extendedprice,
             int64_t* l_quantity,
             int32_t* l_shipdate,
             size_t lineitem_size)
{
  double* d_l_extendedprice;

  hipMalloc(&d_l_extendedprice, sizeof(double) * lineitem_size);

  hipMemcpy(
    d_l_extendedprice, l_extendedprice, sizeof(double) * lineitem_size, hipMemcpyHostToDevice);

  int32_t* d_l_shipdate;

  hipMalloc(&d_l_shipdate, sizeof(int32_t) * lineitem_size);

  hipMemcpy(d_l_shipdate, l_shipdate, sizeof(int32_t) * lineitem_size, hipMemcpyHostToDevice);

  int64_t* d_l_quantity;

  hipMalloc(&d_l_quantity, sizeof(int64_t) * lineitem_size);

  hipMemcpy(d_l_quantity, l_quantity, sizeof(int64_t) * lineitem_size, hipMemcpyHostToDevice);

  double* d_l_discount;

  hipMalloc(&d_l_discount, sizeof(double) * lineitem_size);

  hipMemcpy(d_l_discount, l_discount, sizeof(double) * lineitem_size, hipMemcpyHostToDevice);

  double* d_agg_revenue;

  auto HT0 = cuco::static_map{lineitem_size * 2,
                              cuco::empty_key{(int64_t)-1},
                              cuco::empty_value{(int64_t)-1},
                              thrust::equal_to<int64_t>{},
                              cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};

  auto d_HT0_F = HT0.ref(cuco::find);

  auto d_HT0_I = HT0.ref(cuco::insert);

  pipeline_0<<<std::ceil((float)lineitem_size / (float)32), 32>>>(
    d_l_quantity, d_l_shipdate, d_l_discount, d_l_extendedprice, d_HT0_I, d_HT0_F, lineitem_size);

  auto HT0_size = HT0.size();

  hipMalloc(&d_agg_revenue, sizeof(double) * HT0_size);

  hipMemset(d_agg_revenue, 0, sizeof(double) * HT0_size);

  thrust::device_vector<int64_t> keys_0(HT0_size), vals_0(HT0_size);
  HT0.retrieve_all(keys_0.begin(), vals_0.begin());
  thrust::host_vector<int64_t> h_keys_0(HT0_size);
  thrust::copy(keys_0.begin(), keys_0.end(), h_keys_0.begin());
  thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_0(HT0_size);
  for (int i = 0; i < HT0_size; i++) {
    actual_dict_0[i] = cuco::make_pair(h_keys_0[i], i);
  }
  HT0.clear();
  HT0.insert(actual_dict_0.begin(), actual_dict_0.end());
  pipeline_1<<<std::ceil((float)lineitem_size / (float)32), 32>>>(d_l_extendedprice,
                                                                  d_l_shipdate,
                                                                  d_l_quantity,
                                                                  d_l_discount,
                                                                  d_agg_revenue,
                                                                  d_HT0_I,
                                                                  d_HT0_F,
                                                                  lineitem_size);

  size_t agg_size       = HT0_size;
  double* p_agg_revenue = (double*)malloc(sizeof(double) * agg_size);
  hipMemcpy(p_agg_revenue, d_agg_revenue, sizeof(double) * agg_size, hipMemcpyDeviceToHost);
  for (int i = 0; i < agg_size; i++) {
    std::cout << p_agg_revenue[i] << "\t";
    std::cout << std::endl;
  }
}

int main(int argc, const char** argv)
{
  std::string dbDir         = getDataDir(argv, argc);
  std::string lineitem_file = dbDir + "lineitem.parquet";

  // auto lineitem_table  = getArrowTable(lineitem_file);
  auto lineitem_table  = getArrowTable(lineitem_file);
  size_t lineitem_size = lineitem_table->num_rows();

  auto l_shipdate      = read_column<int32_t>(lineitem_table, "l_shipdate");
  auto l_quantity      = read_column<int64_t>(lineitem_table, "l_quantity");
  auto l_discount      = read_column<double>(lineitem_table, "l_discount");
  auto l_extendedprice = read_column<double>(lineitem_table, "l_extendedprice");

  // for (auto p: c_mktsegment->dict) {
  //   std::cout << p.first << " " << (int)p.second << std::endl;
  // }
  control(
    l_discount.data(), l_extendedprice.data(), l_quantity.data(), l_shipdate.data(), lineitem_size);
}