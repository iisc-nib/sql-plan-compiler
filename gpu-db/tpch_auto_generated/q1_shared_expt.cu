#include "hip/hip_runtime.h"

#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"

#include <cuco/static_map.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iomanip>

template<typename HASHTABLE_INSERT>
__global__ void count_pipeline_5fe3c684fb10(DBDecimalType *lineitem__l_discount,
DBDecimalType *lineitem__l_extendedprice,
DBCharType *lineitem__l_linestatus,
DBCharType *lineitem__l_returnflag,
DBDateType *lineitem__l_shipdate,
DBDecimalType *lineitem__l_tax,
HASHTABLE_INSERT HT_5fe3c6804700,
size_t lineitem_size
) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg__lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg__lineitem__l_shipdate, 10471, Predicate::lte))) return;
auto reg__lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg__lineitem__l_discount = lineitem__l_discount[tid];
auto reg__lineitem__l_tax = lineitem__l_tax[tid];
auto reg__lineitem__l_returnflag = lineitem__l_returnflag[tid];
auto reg__lineitem__l_linestatus = lineitem__l_linestatus[tid];
int64_t KEY_5fe3c6804700 = 0;
KEY_5fe3c6804700 <<= 8;
KEY_5fe3c6804700  |= reg__lineitem__l_returnflag;
KEY_5fe3c6804700 <<= 8;
KEY_5fe3c6804700  |= reg__lineitem__l_linestatus;

HT_5fe3c6804700.insert(cuco::pair{KEY_5fe3c6804700, 1});
return;
}
template<typename HASHTABLE_FIND>
__global__ void main_pipeline_5fe3c684fb10(DBCharType *KEY_5fe3c6804700lineitem__l_linestatus,
DBCharType *KEY_5fe3c6804700lineitem__l_returnflag,
DBDecimalType *aggr0__tmp_attr0,
DBDecimalType *aggr0__tmp_attr1,
DBDecimalType *aggr0__tmp_attr2,
DBDecimalType *aggr0__tmp_attr4,
DBI64Type *aggr0__tmp_attr9,
DBDecimalType *aggr_rw__rw0,
DBI64Type *aggr_rw__rw1,
DBDecimalType *aggr_rw__rw2,
DBI64Type *aggr_rw__rw3,
DBDecimalType *aggr_rw__rw4,
DBI64Type *aggr_rw__rw5,
DBDecimalType *lineitem__l_discount,
DBDecimalType *lineitem__l_extendedprice,
DBCharType *lineitem__l_linestatus,
DBDecimalType *lineitem__l_quantity,
DBCharType *lineitem__l_returnflag,
DBDateType *lineitem__l_shipdate,
DBDecimalType *lineitem__l_tax,
HASHTABLE_FIND HT_5fe3c6804700,
size_t lineitem_size
) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg__lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg__lineitem__l_shipdate, 10471, Predicate::lte))) return;
auto reg__lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg__lineitem__l_discount = lineitem__l_discount[tid];
auto reg__lineitem__l_tax = lineitem__l_tax[tid];
auto reg__lineitem__l_returnflag = lineitem__l_returnflag[tid];
auto reg__lineitem__l_linestatus = lineitem__l_linestatus[tid];
int64_t KEY_5fe3c6804700 = 0;
KEY_5fe3c6804700 <<= 8;
KEY_5fe3c6804700  |= reg__lineitem__l_returnflag;
KEY_5fe3c6804700 <<= 8;
KEY_5fe3c6804700  |= reg__lineitem__l_linestatus;

auto buf_idx_5fe3c6804700 = HT_5fe3c6804700.find(KEY_5fe3c6804700)->second;
KEY_5fe3c6804700lineitem__l_returnflag[buf_idx_5fe3c6804700] = reg__lineitem__l_returnflag;
KEY_5fe3c6804700lineitem__l_linestatus[buf_idx_5fe3c6804700] = reg__lineitem__l_linestatus;
aggregate_sum(&aggr0__tmp_attr9[buf_idx_5fe3c6804700], 1);
auto reg__map0__tmp_attr5 = ((reg__lineitem__l_extendedprice) * ((1) - (reg__lineitem__l_discount))) * ((1) + (reg__lineitem__l_tax));
aggregate_sum(&aggr0__tmp_attr4[buf_idx_5fe3c6804700], reg__map0__tmp_attr5);
auto reg__map0__tmp_attr3 = (reg__lineitem__l_extendedprice) * ((1) - (reg__lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_5fe3c6804700], reg__map0__tmp_attr3);
aggregate_sum(&aggr0__tmp_attr1[buf_idx_5fe3c6804700], reg__lineitem__l_extendedprice);
auto reg__lineitem__l_quantity = lineitem__l_quantity[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5fe3c6804700], reg__lineitem__l_quantity);
aggregate_sum(&aggr_rw__rw0[buf_idx_5fe3c6804700], reg__lineitem__l_discount);
aggregate_sum(&aggr_rw__rw1[buf_idx_5fe3c6804700], 1);
aggregate_sum(&aggr_rw__rw2[buf_idx_5fe3c6804700], reg__lineitem__l_extendedprice);
aggregate_sum(&aggr_rw__rw3[buf_idx_5fe3c6804700], 1);
aggregate_sum(&aggr_rw__rw4[buf_idx_5fe3c6804700], reg__lineitem__l_quantity);
aggregate_sum(&aggr_rw__rw5[buf_idx_5fe3c6804700], 1);
return;
}
__global__ void count_pipeline_5fe3c6857360(DBDecimalType *aggr_rw__rw0,
DBI64Type *aggr_rw__rw1,
DBDecimalType *aggr_rw__rw2,
DBI64Type *aggr_rw__rw3,
DBDecimalType *aggr_rw__rw4,
DBI64Type *aggr_rw__rw5,
size_t MAT_5fe3c6804700_size
) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= MAT_5fe3c6804700_size) return;
auto reg__aggr_rw__rw0 = aggr_rw__rw0[tid];
auto reg__aggr_rw__rw1 = aggr_rw__rw1[tid];
auto reg__aggr_rw__rw2 = aggr_rw__rw2[tid];
auto reg__aggr_rw__rw3 = aggr_rw__rw3[tid];
auto reg__aggr_rw__rw4 = aggr_rw__rw4[tid];
auto reg__aggr_rw__rw5 = aggr_rw__rw5[tid];
}
__global__ void main_pipeline_5fe3c6857360(DBDecimalType *MAT_5fe3c67e4900aggr0__tmp_attr0,
DBDecimalType *MAT_5fe3c67e4900aggr0__tmp_attr1,
DBDecimalType *MAT_5fe3c67e4900aggr0__tmp_attr2,
DBDecimalType *MAT_5fe3c67e4900aggr0__tmp_attr4,
DBDecimalType *MAT_5fe3c67e4900aggr0__tmp_attr6,
DBDecimalType *MAT_5fe3c67e4900aggr0__tmp_attr7,
DBDecimalType *MAT_5fe3c67e4900aggr0__tmp_attr8,
DBI64Type *MAT_5fe3c67e4900aggr0__tmp_attr9,
DBCharType *MAT_5fe3c67e4900lineitem__l_linestatus,
DBCharType *MAT_5fe3c67e4900lineitem__l_returnflag,
DBDecimalType *aggr0__tmp_attr0,
DBDecimalType *aggr0__tmp_attr1,
DBDecimalType *aggr0__tmp_attr2,
DBDecimalType *aggr0__tmp_attr4,
DBI64Type *aggr0__tmp_attr9,
DBDecimalType *aggr_rw__rw0,
DBI64Type *aggr_rw__rw1,
DBDecimalType *aggr_rw__rw2,
DBI64Type *aggr_rw__rw3,
DBDecimalType *aggr_rw__rw4,
DBI64Type *aggr_rw__rw5,
DBCharType *lineitem__l_linestatus,
DBCharType *lineitem__l_returnflag,
size_t MAT_5fe3c6804700_size
) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= MAT_5fe3c6804700_size) return;
auto reg__aggr_rw__rw0 = aggr_rw__rw0[tid];
auto reg__aggr_rw__rw1 = aggr_rw__rw1[tid];
auto reg__aggr_rw__rw2 = aggr_rw__rw2[tid];
auto reg__aggr_rw__rw3 = aggr_rw__rw3[tid];
auto reg__aggr_rw__rw4 = aggr_rw__rw4[tid];
auto reg__aggr_rw__rw5 = aggr_rw__rw5[tid];
auto reg__lineitem__l_returnflag = lineitem__l_returnflag[tid];
MAT_5fe3c67e4900lineitem__l_returnflag[tid] = reg__lineitem__l_returnflag;
auto reg__lineitem__l_linestatus = lineitem__l_linestatus[tid];
MAT_5fe3c67e4900lineitem__l_linestatus[tid] = reg__lineitem__l_linestatus;
auto reg__aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT_5fe3c67e4900aggr0__tmp_attr0[tid] = reg__aggr0__tmp_attr0;
auto reg__aggr0__tmp_attr1 = aggr0__tmp_attr1[tid];
MAT_5fe3c67e4900aggr0__tmp_attr1[tid] = reg__aggr0__tmp_attr1;
auto reg__aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT_5fe3c67e4900aggr0__tmp_attr2[tid] = reg__aggr0__tmp_attr2;
auto reg__aggr0__tmp_attr4 = aggr0__tmp_attr4[tid];
MAT_5fe3c67e4900aggr0__tmp_attr4[tid] = reg__aggr0__tmp_attr4;
auto reg__aggr0__tmp_attr6 = (reg__aggr_rw__rw4) / ((DBDecimalType)(reg__aggr_rw__rw5));
MAT_5fe3c67e4900aggr0__tmp_attr6[tid] = reg__aggr0__tmp_attr6;
auto reg__aggr0__tmp_attr7 = (reg__aggr_rw__rw2) / ((DBDecimalType)(reg__aggr_rw__rw3));
MAT_5fe3c67e4900aggr0__tmp_attr7[tid] = reg__aggr0__tmp_attr7;
auto reg__aggr0__tmp_attr8 = (reg__aggr_rw__rw0) / ((DBDecimalType)(reg__aggr_rw__rw1));
MAT_5fe3c67e4900aggr0__tmp_attr8[tid] = reg__aggr0__tmp_attr8;
auto reg__aggr0__tmp_attr9 = aggr0__tmp_attr9[tid];
MAT_5fe3c67e4900aggr0__tmp_attr9[tid] = reg__aggr0__tmp_attr9;
}

extern "C" void control(
DBI32Type* d_nation__n_nationkey,
DBStringType* d_nation__n_name,
DBI32Type* d_nation__n_regionkey,
DBStringType* d_nation__n_comment,
size_t nation_size,
DBI32Type* d_supplier__s_suppkey,
DBI32Type* d_supplier__s_nationkey,
DBStringType* d_supplier__s_name,
DBStringType* d_supplier__s_address,
DBStringType* d_supplier__s_phone,
DBDecimalType* d_supplier__s_acctbal,
DBStringType* d_supplier__s_comment,
size_t supplier_size,
DBI32Type* d_partsupp__ps_suppkey,
DBI32Type* d_partsupp__ps_partkey,
DBI32Type* d_partsupp__ps_availqty,
DBDecimalType* d_partsupp__ps_supplycost,
DBStringType* d_partsupp__ps_comment,
size_t partsupp_size,
DBI32Type* d_part__p_partkey,
DBStringType* d_part__p_name,
DBStringType* d_part__p_mfgr,
DBStringType* d_part__p_brand,
DBStringType* d_part__p_type,
DBI32Type* d_part__p_size,
DBStringType* d_part__p_container,
DBDecimalType* d_part__p_retailprice,
DBStringType* d_part__p_comment,
size_t part_size,
DBI32Type* d_lineitem__l_orderkey,
DBI32Type* d_lineitem__l_partkey,
DBI32Type* d_lineitem__l_suppkey,
DBI64Type* d_lineitem__l_linenumber,
DBDecimalType* d_lineitem__l_quantity,
DBDecimalType* d_lineitem__l_extendedprice,
DBDecimalType* d_lineitem__l_discount,
DBDecimalType* d_lineitem__l_tax,
DBCharType* d_lineitem__l_returnflag,
DBCharType* d_lineitem__l_linestatus,
DBI32Type* d_lineitem__l_shipdate,
DBI32Type* d_lineitem__l_commitdate,
DBI32Type* d_lineitem__l_receiptdate,
DBStringType* d_lineitem__l_shipinstruct,
DBStringType* d_lineitem__l_shipmode,
DBStringType* d_lineitem__comments,
size_t lineitem_size,
DBI32Type* d_orders__o_orderkey,
DBCharType* d_orders__o_orderstatus,
DBI32Type* d_orders__o_custkey,
DBDecimalType* d_orders__o_totalprice,
DBI32Type* d_orders__o_orderdate,
DBStringType* d_orders__o_orderpriority,
DBStringType* d_orders__o_clerk,
DBI32Type* d_orders__o_shippriority,
DBStringType* d_orders__o_comment,
size_t orders_size,
DBI32Type* d_customer__c_custkey,
DBStringType* d_customer__c_name,
DBStringType* d_customer__c_address,
DBI32Type* d_customer__c_nationkey,
DBStringType* d_customer__c_phone,
DBDecimalType* d_customer__c_acctbal,
DBStringType* d_customer__c_mktsegment,
DBStringType* d_customer__c_comment,
size_t customer_size,
DBI32Type* d_region__r_regionkey,
DBStringType* d_region__r_name,
DBStringType* d_region__r_comment,
size_t region_size
)
{
    auto HT_5fe3c6804700 = cuco::static_map{ 5930889* 2,cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>()};
    count_pipeline_5fe3c684fb10<<<std::ceil((float)lineitem_size/(float)32), 32>>>(d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_linestatus, d_lineitem__l_returnflag, d_lineitem__l_shipdate, d_lineitem__l_tax, HT_5fe3c6804700.ref(cuco::insert), lineitem_size);
    
    thrust::device_vector<int64_t> keys_5fe3c6804700(HT_5fe3c6804700.size()), vals_5fe3c6804700(HT_5fe3c6804700.size());
    HT_5fe3c6804700.retrieve_all(keys_5fe3c6804700.begin(), vals_5fe3c6804700.begin());
    thrust::host_vector<int64_t> h_keys_5fe3c6804700(HT_5fe3c6804700.size());
    thrust::copy(keys_5fe3c6804700.begin(), keys_5fe3c6804700.end(), h_keys_5fe3c6804700.begin());
    thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_5fe3c6804700(HT_5fe3c6804700.size());
    for (int i=0; i < HT_5fe3c6804700.size(); i++) {{
    actual_dict_5fe3c6804700[i] = cuco::make_pair(h_keys_5fe3c6804700[i], i);
    }}
    HT_5fe3c6804700.clear();
    HT_5fe3c6804700.insert(actual_dict_5fe3c6804700.begin(), actual_dict_5fe3c6804700.end());
    DBI64Type*  d_aggr0__tmp_attr9;
    hipMalloc(&d_aggr0__tmp_attr9, sizeof(DBI64Type) * HT_5fe3c6804700.size());
    hipMemset(d_aggr0__tmp_attr9,0 , sizeof(DBI64Type) * HT_5fe3c6804700.size());
    DBDecimalType*  d_aggr0__tmp_attr4;
    hipMalloc(&d_aggr0__tmp_attr4, sizeof(DBDecimalType) * HT_5fe3c6804700.size());
    hipMemset(d_aggr0__tmp_attr4,0 , sizeof(DBDecimalType) * HT_5fe3c6804700.size());
    DBDecimalType*  d_aggr0__tmp_attr2;
    hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * HT_5fe3c6804700.size());
    hipMemset(d_aggr0__tmp_attr2,0 , sizeof(DBDecimalType) * HT_5fe3c6804700.size());
    DBDecimalType*  d_aggr0__tmp_attr1;
    hipMalloc(&d_aggr0__tmp_attr1, sizeof(DBDecimalType) * HT_5fe3c6804700.size());
    hipMemset(d_aggr0__tmp_attr1,0 , sizeof(DBDecimalType) * HT_5fe3c6804700.size());
    DBDecimalType*  d_aggr0__tmp_attr0;
    hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * HT_5fe3c6804700.size());
    hipMemset(d_aggr0__tmp_attr0,0 , sizeof(DBDecimalType) * HT_5fe3c6804700.size());
    DBDecimalType*  d_aggr_rw__rw0;
    hipMalloc(&d_aggr_rw__rw0, sizeof(DBDecimalType) * HT_5fe3c6804700.size());
    hipMemset(d_aggr_rw__rw0,0 , sizeof(DBDecimalType) * HT_5fe3c6804700.size());
    DBI64Type*  d_aggr_rw__rw1;
    hipMalloc(&d_aggr_rw__rw1, sizeof(DBI64Type) * HT_5fe3c6804700.size());
    hipMemset(d_aggr_rw__rw1,0 , sizeof(DBI64Type) * HT_5fe3c6804700.size());
    DBDecimalType*  d_aggr_rw__rw2;
    hipMalloc(&d_aggr_rw__rw2, sizeof(DBDecimalType) * HT_5fe3c6804700.size());
    hipMemset(d_aggr_rw__rw2,0 , sizeof(DBDecimalType) * HT_5fe3c6804700.size());
    DBI64Type*  d_aggr_rw__rw3;
    hipMalloc(&d_aggr_rw__rw3, sizeof(DBI64Type) * HT_5fe3c6804700.size());
    hipMemset(d_aggr_rw__rw3,0 , sizeof(DBI64Type) * HT_5fe3c6804700.size());
    DBDecimalType*  d_aggr_rw__rw4;
    hipMalloc(&d_aggr_rw__rw4, sizeof(DBDecimalType) * HT_5fe3c6804700.size());
    hipMemset(d_aggr_rw__rw4,0 , sizeof(DBDecimalType) * HT_5fe3c6804700.size());
    DBI64Type*  d_aggr_rw__rw5;
    hipMalloc(&d_aggr_rw__rw5, sizeof(DBI64Type) * HT_5fe3c6804700.size());
    hipMemset(d_aggr_rw__rw5,0 , sizeof(DBI64Type) * HT_5fe3c6804700.size());
    auto MAT_5fe3c6804700_size = HT_5fe3c6804700.size();
    DBCharType*  d_KEY_5fe3c6804700lineitem__l_returnflag;
    hipMalloc(&d_KEY_5fe3c6804700lineitem__l_returnflag, sizeof(DBCharType) * HT_5fe3c6804700.size());
    hipMemset(d_KEY_5fe3c6804700lineitem__l_returnflag,0 , sizeof(DBCharType) * HT_5fe3c6804700.size());
    DBCharType*  d_KEY_5fe3c6804700lineitem__l_linestatus;
    hipMalloc(&d_KEY_5fe3c6804700lineitem__l_linestatus, sizeof(DBCharType) * HT_5fe3c6804700.size());
    hipMemset(d_KEY_5fe3c6804700lineitem__l_linestatus,0 , sizeof(DBCharType) * HT_5fe3c6804700.size());
    main_pipeline_5fe3c684fb10<<<std::ceil((float)lineitem_size/(float)32), 32>>>(d_KEY_5fe3c6804700lineitem__l_linestatus, d_KEY_5fe3c6804700lineitem__l_returnflag, d_aggr0__tmp_attr0, d_aggr0__tmp_attr1, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_aggr0__tmp_attr9, d_aggr_rw__rw0, d_aggr_rw__rw1, d_aggr_rw__rw2, d_aggr_rw__rw3, d_aggr_rw__rw4, d_aggr_rw__rw5, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_linestatus, d_lineitem__l_quantity, d_lineitem__l_returnflag, d_lineitem__l_shipdate, d_lineitem__l_tax, HT_5fe3c6804700.ref(cuco::find), lineitem_size);
    
    
    auto MAT_5fe3c67e4900lineitem__l_returnflag = (DBCharType*)malloc(sizeof(DBCharType) * MAT_5fe3c6804700_size);
    DBCharType* d_MAT_5fe3c67e4900lineitem__l_returnflag;
    hipMalloc(&d_MAT_5fe3c67e4900lineitem__l_returnflag, sizeof(DBCharType) * MAT_5fe3c6804700_size);
    auto MAT_5fe3c67e4900lineitem__l_linestatus = (DBCharType*)malloc(sizeof(DBCharType) * MAT_5fe3c6804700_size);
    DBCharType* d_MAT_5fe3c67e4900lineitem__l_linestatus;
    hipMalloc(&d_MAT_5fe3c67e4900lineitem__l_linestatus, sizeof(DBCharType) * MAT_5fe3c6804700_size);
    auto MAT_5fe3c67e4900aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * MAT_5fe3c6804700_size);
    DBDecimalType* d_MAT_5fe3c67e4900aggr0__tmp_attr0;
    hipMalloc(&d_MAT_5fe3c67e4900aggr0__tmp_attr0, sizeof(DBDecimalType) * MAT_5fe3c6804700_size);
    auto MAT_5fe3c67e4900aggr0__tmp_attr1 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * MAT_5fe3c6804700_size);
    DBDecimalType* d_MAT_5fe3c67e4900aggr0__tmp_attr1;
    hipMalloc(&d_MAT_5fe3c67e4900aggr0__tmp_attr1, sizeof(DBDecimalType) * MAT_5fe3c6804700_size);
    auto MAT_5fe3c67e4900aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * MAT_5fe3c6804700_size);
    DBDecimalType* d_MAT_5fe3c67e4900aggr0__tmp_attr2;
    hipMalloc(&d_MAT_5fe3c67e4900aggr0__tmp_attr2, sizeof(DBDecimalType) * MAT_5fe3c6804700_size);
    auto MAT_5fe3c67e4900aggr0__tmp_attr4 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * MAT_5fe3c6804700_size);
    DBDecimalType* d_MAT_5fe3c67e4900aggr0__tmp_attr4;
    hipMalloc(&d_MAT_5fe3c67e4900aggr0__tmp_attr4, sizeof(DBDecimalType) * MAT_5fe3c6804700_size);
    auto MAT_5fe3c67e4900aggr0__tmp_attr6 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * MAT_5fe3c6804700_size);
    DBDecimalType* d_MAT_5fe3c67e4900aggr0__tmp_attr6;
    hipMalloc(&d_MAT_5fe3c67e4900aggr0__tmp_attr6, sizeof(DBDecimalType) * MAT_5fe3c6804700_size);
    auto MAT_5fe3c67e4900aggr0__tmp_attr7 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * MAT_5fe3c6804700_size);
    DBDecimalType* d_MAT_5fe3c67e4900aggr0__tmp_attr7;
    hipMalloc(&d_MAT_5fe3c67e4900aggr0__tmp_attr7, sizeof(DBDecimalType) * MAT_5fe3c6804700_size);
    auto MAT_5fe3c67e4900aggr0__tmp_attr8 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * MAT_5fe3c6804700_size);
    DBDecimalType* d_MAT_5fe3c67e4900aggr0__tmp_attr8;
    hipMalloc(&d_MAT_5fe3c67e4900aggr0__tmp_attr8, sizeof(DBDecimalType) * MAT_5fe3c6804700_size);
    auto MAT_5fe3c67e4900aggr0__tmp_attr9 = (DBI64Type*)malloc(sizeof(DBI64Type) * MAT_5fe3c6804700_size);
    DBI64Type* d_MAT_5fe3c67e4900aggr0__tmp_attr9;
    hipMalloc(&d_MAT_5fe3c67e4900aggr0__tmp_attr9, sizeof(DBI64Type) * MAT_5fe3c6804700_size);
    main_pipeline_5fe3c6857360<<<std::ceil((float)MAT_5fe3c6804700_size/(float)32), 32>>>(d_MAT_5fe3c67e4900aggr0__tmp_attr0, d_MAT_5fe3c67e4900aggr0__tmp_attr1, d_MAT_5fe3c67e4900aggr0__tmp_attr2, d_MAT_5fe3c67e4900aggr0__tmp_attr4, d_MAT_5fe3c67e4900aggr0__tmp_attr6, d_MAT_5fe3c67e4900aggr0__tmp_attr7, d_MAT_5fe3c67e4900aggr0__tmp_attr8, d_MAT_5fe3c67e4900aggr0__tmp_attr9, d_MAT_5fe3c67e4900lineitem__l_linestatus, d_MAT_5fe3c67e4900lineitem__l_returnflag, d_aggr0__tmp_attr0, d_aggr0__tmp_attr1, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_aggr0__tmp_attr9, d_aggr_rw__rw0, d_aggr_rw__rw1, d_aggr_rw__rw2, d_aggr_rw__rw3, d_aggr_rw__rw4, d_aggr_rw__rw5, d_KEY_5fe3c6804700lineitem__l_linestatus, d_KEY_5fe3c6804700lineitem__l_returnflag, MAT_5fe3c6804700_size);
    hipMemcpy(MAT_5fe3c67e4900lineitem__l_returnflag, d_MAT_5fe3c67e4900lineitem__l_returnflag, sizeof(DBCharType) * MAT_5fe3c6804700_size, hipMemcpyDeviceToHost);
    hipMemcpy(MAT_5fe3c67e4900lineitem__l_linestatus, d_MAT_5fe3c67e4900lineitem__l_linestatus, sizeof(DBCharType) * MAT_5fe3c6804700_size, hipMemcpyDeviceToHost);
    hipMemcpy(MAT_5fe3c67e4900aggr0__tmp_attr0, d_MAT_5fe3c67e4900aggr0__tmp_attr0, sizeof(DBDecimalType) * MAT_5fe3c6804700_size, hipMemcpyDeviceToHost);
    hipMemcpy(MAT_5fe3c67e4900aggr0__tmp_attr1, d_MAT_5fe3c67e4900aggr0__tmp_attr1, sizeof(DBDecimalType) * MAT_5fe3c6804700_size, hipMemcpyDeviceToHost);
    hipMemcpy(MAT_5fe3c67e4900aggr0__tmp_attr2, d_MAT_5fe3c67e4900aggr0__tmp_attr2, sizeof(DBDecimalType) * MAT_5fe3c6804700_size, hipMemcpyDeviceToHost);
    hipMemcpy(MAT_5fe3c67e4900aggr0__tmp_attr4, d_MAT_5fe3c67e4900aggr0__tmp_attr4, sizeof(DBDecimalType) * MAT_5fe3c6804700_size, hipMemcpyDeviceToHost);
    hipMemcpy(MAT_5fe3c67e4900aggr0__tmp_attr6, d_MAT_5fe3c67e4900aggr0__tmp_attr6, sizeof(DBDecimalType) * MAT_5fe3c6804700_size, hipMemcpyDeviceToHost);
    hipMemcpy(MAT_5fe3c67e4900aggr0__tmp_attr7, d_MAT_5fe3c67e4900aggr0__tmp_attr7, sizeof(DBDecimalType) * MAT_5fe3c6804700_size, hipMemcpyDeviceToHost);
    hipMemcpy(MAT_5fe3c67e4900aggr0__tmp_attr8, d_MAT_5fe3c67e4900aggr0__tmp_attr8, sizeof(DBDecimalType) * MAT_5fe3c6804700_size, hipMemcpyDeviceToHost);
    hipMemcpy(MAT_5fe3c67e4900aggr0__tmp_attr9, d_MAT_5fe3c67e4900aggr0__tmp_attr9, sizeof(DBI64Type) * MAT_5fe3c6804700_size, hipMemcpyDeviceToHost);
    
    std::cout << std::setprecision(24);
    for (auto i=0ull; i< MAT_5fe3c6804700_size; i++) {
    std::cout << MAT_5fe3c67e4900lineitem__l_returnflag[i] << "\t";
    std::cout << MAT_5fe3c67e4900lineitem__l_linestatus[i] << "\t";
    std::cout << MAT_5fe3c67e4900aggr0__tmp_attr0[i] << "\t";
    std::cout << MAT_5fe3c67e4900aggr0__tmp_attr1[i] << "\t";
    std::cout << MAT_5fe3c67e4900aggr0__tmp_attr2[i] << "\t";
    std::cout << MAT_5fe3c67e4900aggr0__tmp_attr4[i] << "\t";
    std::cout << MAT_5fe3c67e4900aggr0__tmp_attr6[i] << "\t";
    std::cout << MAT_5fe3c67e4900aggr0__tmp_attr7[i] << "\t";
    std::cout << MAT_5fe3c67e4900aggr0__tmp_attr8[i] << "\t";
    std::cout << MAT_5fe3c67e4900aggr0__tmp_attr9[i] << "\t";
    std::cout << std::endl;}
    
}
