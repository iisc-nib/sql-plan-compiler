#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0, buf_idx_0});
BUF_0[buf_idx_0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_FIND>
__global__ void main_3(uint64_t* BUF_0, HASHTABLE_PROBE_PK HT_0, HASHTABLE_FIND HT_2, DBI32Type* KEY_2customer__c_custkey, int* SLOT_COUNT_2, DBI64Type* aggr0__tmp_attr0, DBI32Type* customer__c_custkey, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_0 |= reg_orders__o_custkey;
//Probe Hash table
auto SLOT_0 = HT_0.find(KEY_0);
if (SLOT_0 == HT_0.end()) return;
if (!(true)) return;
uint64_t KEY_2 = 0;
auto reg_customer__c_custkey = customer__c_custkey[BUF_0[SLOT_0->second * 1 + 0]];

KEY_2 |= reg_customer__c_custkey;
//Aggregate in hashtable
auto buf_idx_2 = get_aggregation_slot(KEY_2, HT_2, SLOT_COUNT_2);
auto reg_orders__o_orderkey = orders__o_orderkey[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_2], 1);
KEY_2customer__c_custkey[buf_idx_2] = reg_customer__c_custkey;
}
template<typename HASHTABLE_FIND>
__global__ void main_5(size_t COUNT2, HASHTABLE_FIND HT_4, DBI64Type* KEY_4aggr0__tmp_attr0, int* SLOT_COUNT_4, DBI64Type* aggr0__tmp_attr0, DBI64Type* aggr1__tmp_attr1) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT2) return;
uint64_t KEY_4 = 0;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];

KEY_4 |= (DBI32Type)reg_aggr0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_4 = get_aggregation_slot(KEY_4, HT_4, SLOT_COUNT_4);
aggregate_sum(&aggr1__tmp_attr1[buf_idx_4], 1);
KEY_4aggr0__tmp_attr0[buf_idx_4] = reg_aggr0__tmp_attr0;
}
__global__ void main_7(size_t COUNT4, DBI64Type* MAT6aggr0__tmp_attr0, DBI64Type* MAT6aggr1__tmp_attr1, uint64_t* MAT_IDX6, DBI64Type* aggr0__tmp_attr0, DBI64Type* aggr1__tmp_attr1) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT4) return;
//Materialize buffers
auto mat_idx6 = atomicAdd((int*)MAT_IDX6, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT6aggr0__tmp_attr0[mat_idx6] = reg_aggr0__tmp_attr0;
auto reg_aggr1__tmp_attr1 = aggr1__tmp_attr1[tid];
MAT6aggr1__tmp_attr1[mat_idx6] = reg_aggr1__tmp_attr1;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
size_t COUNT0 = customer_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_customer__c_custkey, customer_size);
hipFree(d_BUF_IDX_0);
size_t COUNT2 = 15000000;
auto d_HT_2 = cuco::static_map{ (int)15000000*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_2;
hipMalloc(&d_SLOT_COUNT_2, sizeof(int));
hipMemset(d_SLOT_COUNT_2, 0, sizeof(int));
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT2);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT2);
DBI32Type* d_KEY_2customer__c_custkey;
hipMalloc(&d_KEY_2customer__c_custkey, sizeof(DBI32Type) * COUNT2);
hipMemset(d_KEY_2customer__c_custkey, 0, sizeof(DBI32Type) * COUNT2);
main_3<<<std::ceil((float)orders_size/128.), 128>>>(d_BUF_0, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::insert_and_find), d_KEY_2customer__c_custkey, d_SLOT_COUNT_2, d_aggr0__tmp_attr0, d_customer__c_custkey, d_orders__o_custkey, d_orders__o_orderkey, orders_size);
COUNT2 = d_HT_2.size();
size_t COUNT4 = 15000000;
auto d_HT_4 = cuco::static_map{ (int)15000000*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_4;
hipMalloc(&d_SLOT_COUNT_4, sizeof(int));
hipMemset(d_SLOT_COUNT_4, 0, sizeof(int));
//Aggregate in hashtable
DBI64Type* d_aggr1__tmp_attr1;
hipMalloc(&d_aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT4);
hipMemset(d_aggr1__tmp_attr1, 0, sizeof(DBI64Type) * COUNT4);
DBI64Type* d_KEY_4aggr0__tmp_attr0;
hipMalloc(&d_KEY_4aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT4);
hipMemset(d_KEY_4aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT4);
main_5<<<std::ceil((float)COUNT2/128.), 128>>>(COUNT2, d_HT_4.ref(cuco::insert_and_find), d_KEY_4aggr0__tmp_attr0, d_SLOT_COUNT_4, d_aggr0__tmp_attr0, d_aggr1__tmp_attr1);
COUNT4 = d_HT_4.size();
size_t COUNT6 = COUNT4;
//Materialize buffers
uint64_t* d_MAT_IDX6;
hipMalloc(&d_MAT_IDX6, sizeof(uint64_t));
hipMemset(d_MAT_IDX6, 0, sizeof(uint64_t));
auto MAT6aggr0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT6);
DBI64Type* d_MAT6aggr0__tmp_attr0;
hipMalloc(&d_MAT6aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT6);
auto MAT6aggr1__tmp_attr1 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT6);
DBI64Type* d_MAT6aggr1__tmp_attr1;
hipMalloc(&d_MAT6aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT6);
main_7<<<std::ceil((float)COUNT4/128.), 128>>>(COUNT4, d_MAT6aggr0__tmp_attr0, d_MAT6aggr1__tmp_attr1, d_MAT_IDX6, d_KEY_4aggr0__tmp_attr0, d_aggr1__tmp_attr1);
uint64_t MATCOUNT_6 = 0;
hipMemcpy(&MATCOUNT_6, d_MAT_IDX6, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT6aggr0__tmp_attr0, d_MAT6aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT6, hipMemcpyDeviceToHost);
hipMemcpy(MAT6aggr1__tmp_attr1, d_MAT6aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT6, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < MATCOUNT_6; i++) { std::cout << "" << MAT6aggr0__tmp_attr0[i];
std::cout << "|" << MAT6aggr1__tmp_attr1[i];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_KEY_2customer__c_custkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_KEY_4aggr0__tmp_attr0);
hipFree(d_aggr1__tmp_attr1);
hipFree(d_MAT6aggr0__tmp_attr0);
hipFree(d_MAT6aggr1__tmp_attr1);
hipFree(d_MAT_IDX6);
free(MAT6aggr0__tmp_attr0);
free(MAT6aggr1__tmp_attr1);
}