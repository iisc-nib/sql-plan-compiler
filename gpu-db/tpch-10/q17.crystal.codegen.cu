#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_1(uint64_t* COUNT0, DBStringType* part__p_brand, DBStringType* part__p_container, size_t part_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_part__p_container[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_container[ITEM] = part__p_container[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq);
}
DBStringType reg_part__p_brand[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_brand[ITEM] = part__p_brand[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT0, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBStringType* part__p_brand, DBStringType* part__p_container, DBI32Type* part__p_partkey, size_t part_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_part__p_container[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_container[ITEM] = part__p_container[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_part__p_container[ITEM], "MED BOX", Predicate::eq);
}
DBStringType reg_part__p_brand[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_brand[ITEM] = part__p_brand[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_part__p_brand[ITEM], "Brand#23", Predicate::eq);
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0[ITEM], buf_idx_0});
BUF_0[(buf_idx_0) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_3(uint64_t* BUF_0, HASHTABLE_PROBE HT_0, HASHTABLE_INSERT HT_2, size_t lineitem_size, DBI32Type* lineitem_u_1__l_partkey) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_lineitem_u_1__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem_u_1__l_partkey[ITEM] = lineitem_u_1__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_lineitem_u_1__l_partkey[ITEM];
}
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_lineitem_u_1__l_partkey[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_2.insert(cuco::pair{KEY_2[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_3(uint64_t* BUF_0, HASHTABLE_PROBE HT_0, HASHTABLE_FIND HT_2, DBI32Type* KEY_2lineitem_u_1__l_partkey, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, size_t lineitem_size, DBI32Type* lineitem_u_1__l_partkey, DBDecimalType* lineitem_u_1__l_quantity, DBI32Type* moved_aggr__p_partkey, DBI32Type* part__p_partkey) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_lineitem_u_1__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem_u_1__l_partkey[ITEM] = lineitem_u_1__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_lineitem_u_1__l_partkey[ITEM];
}
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_lineitem_u_1__l_partkey[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem_u_1__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem_u_1__l_quantity[ITEM] = lineitem_u_1__l_quantity[ITEM*TB + tid];
}
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_part__p_partkey[ITEM] = part__p_partkey[BUF_0[slot_second0[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_2 = HT_2.find(KEY_2[ITEM])->second;
aggregate_sum(&aggr_rw__rw0[buf_idx_2], reg_lineitem_u_1__l_quantity[ITEM]);
aggregate_sum(&aggr_rw__rw1[buf_idx_2], 1);
aggregate_any(&moved_aggr__p_partkey[buf_idx_2], reg_part__p_partkey[ITEM]);
KEY_2lineitem_u_1__l_partkey[buf_idx_2] = reg_lineitem_u_1__l_partkey[ITEM];
}
}
__global__ void count_5(size_t COUNT2, uint64_t* COUNT4, DBI32Type* moved_aggr__p_partkey) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT4, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_4, uint64_t* BUF_IDX_4, size_t COUNT2, HASHTABLE_INSERT HT_4, DBI32Type* moved_aggr__p_partkey) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
reg_part__p_partkey[ITEM] = moved_aggr__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT2); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_4 = atomicAdd((int*)BUF_IDX_4, 1);
HT_4.insert(cuco::pair{KEY_4[ITEM], buf_idx_4});
BUF_4[(buf_idx_4) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_7(uint64_t* BUF_4, HASHTABLE_PROBE HT_4, HASHTABLE_INSERT HT_6, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, size_t lineitem_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
int64_t slot_second4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0; continue;}
slot_second4[ITEM] = SLOT_4->second;
}
DBDecimalType reg_lineitem__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_quantity[ITEM] = lineitem__l_quantity[ITEM*TB + tid];
}
DBI64Type reg_aggr_rw__rw1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr_rw__rw1[ITEM] = aggr_rw__rw1[BUF_4[slot_second4[ITEM] * 1 + 0]];
}
DBDecimalType reg_aggr_rw__rw0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr_rw__rw0[ITEM] = aggr_rw__rw0[BUF_4[slot_second4[ITEM] * 1 + 0]];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr0__tmp_attr0[ITEM] = (reg_aggr_rw__rw0[ITEM]) / ((DBDecimalType)(reg_aggr_rw__rw1[ITEM]));
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (0.2) * (reg_aggr0__tmp_attr0[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(((DBDecimalType)reg_lineitem__l_quantity[ITEM]), reg_map0__tmp_attr1[ITEM], Predicate::lt)) && (true);
}
uint64_t KEY_6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_6.insert(cuco::pair{KEY_6[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_7(uint64_t* BUF_4, HASHTABLE_PROBE HT_4, HASHTABLE_FIND HT_6, DBDecimalType* aggr1__tmp_attr2, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, size_t lineitem_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
int64_t slot_second4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0; continue;}
slot_second4[ITEM] = SLOT_4->second;
}
DBDecimalType reg_lineitem__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_quantity[ITEM] = lineitem__l_quantity[ITEM*TB + tid];
}
DBI64Type reg_aggr_rw__rw1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr_rw__rw1[ITEM] = aggr_rw__rw1[BUF_4[slot_second4[ITEM] * 1 + 0]];
}
DBDecimalType reg_aggr_rw__rw0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr_rw__rw0[ITEM] = aggr_rw__rw0[BUF_4[slot_second4[ITEM] * 1 + 0]];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_aggr0__tmp_attr0[ITEM] = (reg_aggr_rw__rw0[ITEM]) / ((DBDecimalType)(reg_aggr_rw__rw1[ITEM]));
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (0.2) * (reg_aggr0__tmp_attr0[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= (evaluatePredicate(((DBDecimalType)reg_lineitem__l_quantity[ITEM]), reg_map0__tmp_attr1[ITEM], Predicate::lt)) && (true);
}
uint64_t KEY_6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_6 = HT_6.find(KEY_6[ITEM])->second;
aggregate_sum(&aggr1__tmp_attr2[buf_idx_6], reg_lineitem__l_extendedprice[ITEM]);
}
}
__global__ void count_9(size_t COUNT6, uint64_t* COUNT8) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT6); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT8, 1);
}
}
__global__ void main_9(size_t COUNT6, DBDecimalType* MAT8map1__tmp_attr3, uint64_t* MAT_IDX8, DBDecimalType* aggr1__tmp_attr2) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize buffers
DBDecimalType reg_aggr1__tmp_attr2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT6); ++ITEM) {
reg_aggr1__tmp_attr2[ITEM] = aggr1__tmp_attr2[ITEM*TB + tid];
}
DBDecimalType reg_map1__tmp_attr3[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT6); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map1__tmp_attr3[ITEM] = (reg_aggr1__tmp_attr2[ITEM]) / (7.0);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT6); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx8 = atomicAdd((int*)MAT_IDX8, 1);
MAT8map1__tmp_attr3[mat_idx8] = reg_map1__tmp_attr3[ITEM];
}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)part_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT0, d_part__p_brand, d_part__p_container, part_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)part_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_part__p_brand, d_part__p_container, d_part__p_partkey, part_size);
//Create aggregation hash table
auto d_HT_2 = cuco::static_map{ (int)59986052*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_3<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::insert), lineitem_size, d_lineitem__l_partkey);
size_t COUNT2 = d_HT_2.size();
thrust::device_vector<int64_t> keys_2(COUNT2), vals_2(COUNT2);
d_HT_2.retrieve_all(keys_2.begin(), vals_2.begin());
d_HT_2.clear();
int64_t* raw_keys2 = thrust::raw_pointer_cast(keys_2.data());
insertKeys<<<std::ceil((float)COUNT2/128.), 128>>>(raw_keys2, d_HT_2.ref(cuco::insert), COUNT2);
//Aggregate in hashtable
DBDecimalType* d_aggr_rw__rw0;
hipMalloc(&d_aggr_rw__rw0, sizeof(DBDecimalType) * COUNT2);
hipMemset(d_aggr_rw__rw0, 0, sizeof(DBDecimalType) * COUNT2);
DBI64Type* d_aggr_rw__rw1;
hipMalloc(&d_aggr_rw__rw1, sizeof(DBI64Type) * COUNT2);
hipMemset(d_aggr_rw__rw1, 0, sizeof(DBI64Type) * COUNT2);
DBI32Type* d_moved_aggr__p_partkey;
hipMalloc(&d_moved_aggr__p_partkey, sizeof(DBI32Type) * COUNT2);
hipMemset(d_moved_aggr__p_partkey, 0, sizeof(DBI32Type) * COUNT2);
DBI32Type* d_KEY_2lineitem_u_1__l_partkey;
hipMalloc(&d_KEY_2lineitem_u_1__l_partkey, sizeof(DBI32Type) * COUNT2);
hipMemset(d_KEY_2lineitem_u_1__l_partkey, 0, sizeof(DBI32Type) * COUNT2);
main_3<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_0, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::find), d_KEY_2lineitem_u_1__l_partkey, d_aggr_rw__rw0, d_aggr_rw__rw1, lineitem_size, d_lineitem__l_partkey, d_lineitem__l_quantity, d_moved_aggr__p_partkey, d_part__p_partkey);
//Materialize count
uint64_t* d_COUNT4;
hipMalloc(&d_COUNT4, sizeof(uint64_t));
hipMemset(d_COUNT4, 0, sizeof(uint64_t));
count_5<<<std::ceil((float)COUNT2/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT2, d_COUNT4, d_moved_aggr__p_partkey);
uint64_t COUNT4;
hipMemcpy(&COUNT4, d_COUNT4, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 1);
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)COUNT2/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_BUF_IDX_4, COUNT2, d_HT_4.ref(cuco::insert), d_moved_aggr__p_partkey);
//Create aggregation hash table
auto d_HT_6 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_7<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::insert), d_aggr_rw__rw0, d_aggr_rw__rw1, d_lineitem__l_partkey, d_lineitem__l_quantity, lineitem_size);
size_t COUNT6 = d_HT_6.size();
thrust::device_vector<int64_t> keys_6(COUNT6), vals_6(COUNT6);
d_HT_6.retrieve_all(keys_6.begin(), vals_6.begin());
d_HT_6.clear();
int64_t* raw_keys6 = thrust::raw_pointer_cast(keys_6.data());
insertKeys<<<std::ceil((float)COUNT6/128.), 128>>>(raw_keys6, d_HT_6.ref(cuco::insert), COUNT6);
//Aggregate in hashtable
DBDecimalType* d_aggr1__tmp_attr2;
hipMalloc(&d_aggr1__tmp_attr2, sizeof(DBDecimalType) * COUNT6);
hipMemset(d_aggr1__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT6);
main_7<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_4, d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::find), d_aggr1__tmp_attr2, d_aggr_rw__rw0, d_aggr_rw__rw1, d_lineitem__l_extendedprice, d_lineitem__l_partkey, d_lineitem__l_quantity, lineitem_size);
//Materialize count
uint64_t* d_COUNT8;
hipMalloc(&d_COUNT8, sizeof(uint64_t));
hipMemset(d_COUNT8, 0, sizeof(uint64_t));
count_9<<<std::ceil((float)COUNT6/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT6, d_COUNT8);
uint64_t COUNT8;
hipMemcpy(&COUNT8, d_COUNT8, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX8;
hipMalloc(&d_MAT_IDX8, sizeof(uint64_t));
hipMemset(d_MAT_IDX8, 0, sizeof(uint64_t));
auto MAT8map1__tmp_attr3 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT8);
DBDecimalType* d_MAT8map1__tmp_attr3;
hipMalloc(&d_MAT8map1__tmp_attr3, sizeof(DBDecimalType) * COUNT8);
main_9<<<std::ceil((float)COUNT6/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT6, d_MAT8map1__tmp_attr3, d_MAT_IDX8, d_aggr1__tmp_attr2);
hipMemcpy(MAT8map1__tmp_attr3, d_MAT8map1__tmp_attr3, sizeof(DBDecimalType) * COUNT8, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < COUNT8; i++) { std::cout << "" << MAT8map1__tmp_attr3[i];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
      size_t aux_mem = usedGpuMem() - used_mem;
      std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_COUNT0);
hipFree(d_KEY_2lineitem_u_1__l_partkey);
hipFree(d_aggr_rw__rw0);
hipFree(d_aggr_rw__rw1);
hipFree(d_moved_aggr__p_partkey);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_COUNT4);
hipFree(d_aggr1__tmp_attr2);
hipFree(d_COUNT8);
hipFree(d_MAT8map1__tmp_attr3);
hipFree(d_MAT_IDX8);
free(MAT8map1__tmp_attr3);
}