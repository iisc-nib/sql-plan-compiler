#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBI32Type* part__p_partkey, DBStringType* part__p_type, size_t part_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_part__p_type[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_type[ITEM] = part__p_type[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_part__p_type[ITEM], "ECONOMY ANODIZED STEEL", Predicate::eq);
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_0.insert(cuco::pair{KEY_0[ITEM], ITEM*TB + tid});
BUF_0[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_INSERT HT_2, DBStringType* region__r_name, DBI32Type* region__r_regionkey, size_t region_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_region__r_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
reg_region__r_name[ITEM] = region__r_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_region__r_name[ITEM], "AMERICA", Predicate::eq);
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_region__r_regionkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
reg_region__r_regionkey[ITEM] = region__r_regionkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_region__r_regionkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < region_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_2.insert(cuco::pair{KEY_2[ITEM], ITEM*TB + tid});
BUF_2[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_5(uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_INSERT HT_4, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_orders__o_orderdate[ITEM], 9131, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate[ITEM], 9861, Predicate::lte);
}
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
reg_orders__o_orderkey[ITEM] = orders__o_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_orders__o_orderkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < orders_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_4.insert(cuco::pair{KEY_4[ITEM], ITEM*TB + tid});
BUF_4[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_7(uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_INSERT HT_6, DBI32Type* customer__c_custkey, size_t customer_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
reg_customer__c_custkey[ITEM] = customer__c_custkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_customer__c_custkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < customer_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_6.insert(cuco::pair{KEY_6[ITEM], ITEM*TB + tid});
BUF_6[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_9(uint64_t* BUF_2, uint64_t* BUF_8, uint64_t* BUF_IDX_8, HASHTABLE_PROBE HT_2, HASHTABLE_INSERT HT_8, DBI32Type* n1___n_nationkey, DBI32Type* n1___n_regionkey, size_t nation_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_n1___n_regionkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n1___n_regionkey[ITEM] = n1___n_regionkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_n1___n_regionkey[ITEM];
}
//Probe Hash table
int64_t slot_second2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_2 = HT_2.find(KEY_2[ITEM]);
if (SLOT_2 == HT_2.end()) {selection_flags[ITEM] = 0; continue;}
slot_second2[ITEM] = SLOT_2->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_n1___n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n1___n_nationkey[ITEM] = n1___n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_n1___n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_8.insert(cuco::pair{KEY_8[ITEM], ITEM*TB + tid});
BUF_8[(ITEM*TB + tid) * 2 + 0] = BUF_2[slot_second2[ITEM] * 1 + 0];
BUF_8[(ITEM*TB + tid) * 2 + 1] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_11(uint64_t* BUF_10, uint64_t* BUF_IDX_10, HASHTABLE_INSERT HT_10, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_10.insert(cuco::pair{KEY_10[ITEM], ITEM*TB + tid});
BUF_10[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_13(uint64_t* BUF_12, uint64_t* BUF_IDX_12, HASHTABLE_INSERT HT_12, DBI32Type* n2___n_nationkey, size_t nation_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_12[ITEMS_PER_THREAD];
DBI32Type reg_n2___n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_n2___n_nationkey[ITEM] = n2___n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_12[ITEM] = 0;
KEY_12[ITEM] |= reg_n2___n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_12.insert(cuco::pair{KEY_12[ITEM], ITEM*TB + tid});
BUF_12[(ITEM*TB + tid) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_15(uint64_t* BUF_0, uint64_t* BUF_10, uint64_t* BUF_12, uint64_t* BUF_4, uint64_t* BUF_6, uint64_t* BUF_8, HASHTABLE_PROBE HT_0, HASHTABLE_PROBE HT_10, HASHTABLE_PROBE HT_12, HASHTABLE_FIND HT_14, HASHTABLE_PROBE HT_4, HASHTABLE_PROBE HT_6, HASHTABLE_PROBE HT_8, DBI64Type* KEY_14map0__tmp_attr0, int* SLOT_COUNT_14, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, DBI32Type* customer__c_nationkey, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n2___n_name, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* supplier__s_nationkey) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
//Probe Hash table
int64_t slot_second0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0; continue;}
slot_second0[ITEM] = SLOT_0->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_4[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_orderkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_orderkey[ITEM] = lineitem__l_orderkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_lineitem__l_orderkey[ITEM];
}
//Probe Hash table
int64_t slot_second4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_4 = HT_4.find(KEY_4[ITEM]);
if (SLOT_4 == HT_4.end()) {selection_flags[ITEM] = 0; continue;}
slot_second4[ITEM] = SLOT_4->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_orders__o_custkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_custkey[ITEM] = orders__o_custkey[BUF_4[slot_second4[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_orders__o_custkey[ITEM];
}
//Probe Hash table
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_customer__c_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_customer__c_nationkey[ITEM] = customer__c_nationkey[BUF_6[slot_second6[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_customer__c_nationkey[ITEM];
}
//Probe Hash table
int64_t slot_second8[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_8 = HT_8.find(KEY_8[ITEM]);
if (SLOT_8 == HT_8.end()) {selection_flags[ITEM] = 0; continue;}
slot_second8[ITEM] = SLOT_8->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_10[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_suppkey[ITEM] = lineitem__l_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_10[ITEM] = 0;
KEY_10[ITEM] |= reg_lineitem__l_suppkey[ITEM];
}
//Probe Hash table
int64_t slot_second10[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_10 = HT_10.find(KEY_10[ITEM]);
if (SLOT_10 == HT_10.end()) {selection_flags[ITEM] = 0; continue;}
slot_second10[ITEM] = SLOT_10->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_12[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_supplier__s_nationkey[ITEM] = supplier__s_nationkey[BUF_10[slot_second10[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_12[ITEM] = 0;
KEY_12[ITEM] |= reg_supplier__s_nationkey[ITEM];
}
//Probe Hash table
int64_t slot_second12[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_12 = HT_12.find(KEY_12[ITEM]);
if (SLOT_12 == HT_12.end()) {selection_flags[ITEM] = 0; continue;}
slot_second12[ITEM] = SLOT_12->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
DBDateType reg_orders__o_orderdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_orders__o_orderdate[ITEM] = orders__o_orderdate[BUF_4[slot_second4[ITEM] * 1 + 0]];
}
uint64_t KEY_14[ITEMS_PER_THREAD];
DBI64Type reg_map0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr0[ITEM] = ExtractFromDate("year", reg_orders__o_orderdate[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_14[ITEM] = 0;
KEY_14[ITEM] |= (DBI32Type)reg_map0__tmp_attr0[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_discount[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_discount[ITEM] = lineitem__l_discount[ITEM*TB + tid];
}
DBDecimalType reg_lineitem__l_extendedprice[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_extendedprice[ITEM] = lineitem__l_extendedprice[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (reg_lineitem__l_extendedprice[ITEM]) * ((1.0) - (reg_lineitem__l_discount[ITEM]));
}
DBStringType reg_n2___n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_n2___n_name[ITEM] = n2___n_name[BUF_12[slot_second12[ITEM] * 1 + 0]];
}
DBDecimalType reg_map1__tmp_attr3[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map1__tmp_attr3[ITEM] = ((evaluatePredicate(reg_n2___n_name[ITEM], "BRAZIL", Predicate::eq))) ? ((reg_map0__tmp_attr1[ITEM])) : ((0.0));
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_14 = get_aggregation_slot(KEY_14[ITEM], HT_14, SLOT_COUNT_14);
aggregate_sum(&aggr0__tmp_attr4[buf_idx_14], reg_map0__tmp_attr1[ITEM]);
aggregate_sum(&aggr0__tmp_attr2[buf_idx_14], reg_map1__tmp_attr3[ITEM]);
KEY_14map0__tmp_attr0[buf_idx_14] = reg_map0__tmp_attr0[ITEM];
}
}
__global__ void main_17(size_t COUNT14, DBI64Type* MAT16map0__tmp_attr0, DBDecimalType* MAT16map2__tmp_attr5, uint64_t* MAT_IDX16, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, DBI64Type* map0__tmp_attr0) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
//Materialize buffers
DBI64Type reg_map0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT14); ++ITEM) {
reg_map0__tmp_attr0[ITEM] = map0__tmp_attr0[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT14); ++ITEM) {
reg_aggr0__tmp_attr4[ITEM] = aggr0__tmp_attr4[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT14); ++ITEM) {
reg_aggr0__tmp_attr2[ITEM] = aggr0__tmp_attr2[ITEM*TB + tid];
}
DBDecimalType reg_map2__tmp_attr5[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT14); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map2__tmp_attr5[ITEM] = (reg_aggr0__tmp_attr2[ITEM]) / (reg_aggr0__tmp_attr4[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT14); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx16 = atomicAdd((int*)MAT_IDX16, 1);
MAT16map0__tmp_attr0[mat_idx16] = reg_map0__tmp_attr0[ITEM];
MAT16map2__tmp_attr5[mat_idx16] = reg_map2__tmp_attr5[ITEM];
}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
size_t COUNT0 = part_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)part_size/(float)TILE_SIZE), TB>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_part__p_partkey, d_part__p_type, part_size);
size_t COUNT2 = region_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 1);
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)region_size/(float)TILE_SIZE), TB>>>(d_BUF_2, d_BUF_IDX_2, d_HT_2.ref(cuco::insert), d_region__r_name, d_region__r_regionkey, region_size);
size_t COUNT4 = orders_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 1);
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)orders_size/(float)TILE_SIZE), TB>>>(d_BUF_4, d_BUF_IDX_4, d_HT_4.ref(cuco::insert), d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
size_t COUNT6 = customer_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 1);
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)customer_size/(float)TILE_SIZE), TB>>>(d_BUF_6, d_BUF_IDX_6, d_HT_6.ref(cuco::insert), d_customer__c_custkey, customer_size);
size_t COUNT8 = nation_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_8;
hipMalloc(&d_BUF_IDX_8, sizeof(uint64_t));
hipMemset(d_BUF_IDX_8, 0, sizeof(uint64_t));
uint64_t* d_BUF_8;
hipMalloc(&d_BUF_8, sizeof(uint64_t) * COUNT8 * 2);
auto d_HT_8 = cuco::static_map{ (int)COUNT8*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_9<<<std::ceil((float)nation_size/(float)TILE_SIZE), TB>>>(d_BUF_2, d_BUF_8, d_BUF_IDX_8, d_HT_2.ref(cuco::find), d_HT_8.ref(cuco::insert), d_nation__n_nationkey, d_nation__n_regionkey, nation_size);
size_t COUNT10 = supplier_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_10;
hipMalloc(&d_BUF_IDX_10, sizeof(uint64_t));
hipMemset(d_BUF_IDX_10, 0, sizeof(uint64_t));
uint64_t* d_BUF_10;
hipMalloc(&d_BUF_10, sizeof(uint64_t) * COUNT10 * 1);
auto d_HT_10 = cuco::static_map{ (int)COUNT10*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_11<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TB>>>(d_BUF_10, d_BUF_IDX_10, d_HT_10.ref(cuco::insert), d_supplier__s_suppkey, supplier_size);
size_t COUNT12 = nation_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_12;
hipMalloc(&d_BUF_IDX_12, sizeof(uint64_t));
hipMemset(d_BUF_IDX_12, 0, sizeof(uint64_t));
uint64_t* d_BUF_12;
hipMalloc(&d_BUF_12, sizeof(uint64_t) * COUNT12 * 1);
auto d_HT_12 = cuco::static_map{ (int)COUNT12*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_13<<<std::ceil((float)nation_size/(float)TILE_SIZE), TB>>>(d_BUF_12, d_BUF_IDX_12, d_HT_12.ref(cuco::insert), d_nation__n_nationkey, nation_size);
size_t COUNT14 = 37872;
auto d_HT_14 = cuco::static_map{ (int)37872*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_14;
hipMalloc(&d_SLOT_COUNT_14, sizeof(int));
hipMemset(d_SLOT_COUNT_14, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr4;
hipMalloc(&d_aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT14);
hipMemset(d_aggr0__tmp_attr4, 0, sizeof(DBDecimalType) * COUNT14);
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT14);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT14);
DBI64Type* d_KEY_14map0__tmp_attr0;
hipMalloc(&d_KEY_14map0__tmp_attr0, sizeof(DBI64Type) * COUNT14);
hipMemset(d_KEY_14map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT14);
main_15<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TB>>>(d_BUF_0, d_BUF_10, d_BUF_12, d_BUF_4, d_BUF_6, d_BUF_8, d_HT_0.ref(cuco::find), d_HT_10.ref(cuco::find), d_HT_12.ref(cuco::find), d_HT_14.ref(cuco::insert_and_find), d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::find), d_KEY_14map0__tmp_attr0, d_SLOT_COUNT_14, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_customer__c_nationkey, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_orders__o_custkey, d_orders__o_orderdate, d_supplier__s_nationkey);
COUNT14 = d_HT_14.size();
size_t COUNT16 = COUNT14;
//Materialize buffers
uint64_t* d_MAT_IDX16;
hipMalloc(&d_MAT_IDX16, sizeof(uint64_t));
hipMemset(d_MAT_IDX16, 0, sizeof(uint64_t));
auto MAT16map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT16);
DBI64Type* d_MAT16map0__tmp_attr0;
hipMalloc(&d_MAT16map0__tmp_attr0, sizeof(DBI64Type) * COUNT16);
auto MAT16map2__tmp_attr5 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT16);
DBDecimalType* d_MAT16map2__tmp_attr5;
hipMalloc(&d_MAT16map2__tmp_attr5, sizeof(DBDecimalType) * COUNT16);
main_17<<<std::ceil((float)COUNT14/(float)TILE_SIZE), TB>>>(COUNT14, d_MAT16map0__tmp_attr0, d_MAT16map2__tmp_attr5, d_MAT_IDX16, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_KEY_14map0__tmp_attr0);
uint64_t MATCOUNT_16 = 0;
hipMemcpy(&MATCOUNT_16, d_MAT_IDX16, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT16map0__tmp_attr0, d_MAT16map0__tmp_attr0, sizeof(DBI64Type) * COUNT16, hipMemcpyDeviceToHost);
hipMemcpy(MAT16map2__tmp_attr5, d_MAT16map2__tmp_attr5, sizeof(DBDecimalType) * COUNT16, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < MATCOUNT_16; i++) { std::cout << "" << MAT16map0__tmp_attr0[i];
std::cout << "|" << MAT16map2__tmp_attr5[i];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_BUF_8);
hipFree(d_BUF_IDX_8);
hipFree(d_BUF_10);
hipFree(d_BUF_IDX_10);
hipFree(d_BUF_12);
hipFree(d_BUF_IDX_12);
hipFree(d_KEY_14map0__tmp_attr0);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_aggr0__tmp_attr4);
hipFree(d_MAT16map0__tmp_attr0);
hipFree(d_MAT16map2__tmp_attr5);
hipFree(d_MAT_IDX16);
free(MAT16map0__tmp_attr0);
free(MAT16map2__tmp_attr5);
}