#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
#define ITEMS_PER_THREAD 4
#define TILE_SIZE 512
#define TB TILE_SIZE/ITEMS_PER_THREAD
__global__ void count_1(uint64_t* COUNT0, DBStringType* part__p_name, size_t part_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_part__p_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_name[ITEM] = part__p_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= Like(reg_part__p_name[ITEM], "forest", "", nullptr, nullptr, 0);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT0, 1);
}
}
template<typename HASHTABLE_INSERT_SJ>
__global__ void main_1(HASHTABLE_INSERT_SJ HT_0, DBStringType* part__p_name, DBI32Type* part__p_partkey, size_t part_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_part__p_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_name[ITEM] = part__p_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= Like(reg_part__p_name[ITEM], "forest", "", nullptr, nullptr, 0);
}
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_part__p_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
reg_part__p_partkey[ITEM] = part__p_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_part__p_partkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < part_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_0.insert(cuco::pair{KEY_0[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE_SJ>
__global__ void count_3(uint64_t* COUNT2, HASHTABLE_PROBE_SJ HT_0, DBI32Type* partsupp__ps_partkey, size_t partsupp_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_partsupp__ps_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
reg_partsupp__ps_partkey[ITEM] = partsupp__ps_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_partsupp__ps_partkey[ITEM];
}
//Probe Hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0;}
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT2, 1);
}
}
template<typename HASHTABLE_PROBE_SJ, typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_PROBE_SJ HT_0, HASHTABLE_INSERT HT_2, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_0[ITEMS_PER_THREAD];
DBI32Type reg_partsupp__ps_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
reg_partsupp__ps_partkey[ITEM] = partsupp__ps_partkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_0[ITEM] = 0;
KEY_0[ITEM] |= reg_partsupp__ps_partkey[ITEM];
}
//Probe Hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_0 = HT_0.find(KEY_0[ITEM]);
if (SLOT_0 == HT_0.end()) {selection_flags[ITEM] = 0;}
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_partsupp__ps_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
reg_partsupp__ps_suppkey[ITEM] = partsupp__ps_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_partsupp__ps_partkey[ITEM];
KEY_2[ITEM] <<= 32;
KEY_2[ITEM] |= reg_partsupp__ps_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < partsupp_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_2 = atomicAdd((int*)BUF_IDX_2, 1);
HT_2.insert(cuco::pair{KEY_2[ITEM], buf_idx_2});
BUF_2[(buf_idx_2) * 1 + 0] = ITEM*TB + tid;
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void count_5(uint64_t* BUF_2, HASHTABLE_PROBE HT_2, HASHTABLE_INSERT HT_4, DBI32Type* lineitem__l_partkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_lineitem__l_shipdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipdate[ITEM] = lineitem__l_shipdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 9131, Predicate::lt);
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
DBI32Type reg_lineitem__l_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_suppkey[ITEM] = lineitem__l_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_lineitem__l_partkey[ITEM];
KEY_2[ITEM] <<= 32;
KEY_2[ITEM] |= reg_lineitem__l_suppkey[ITEM];
}
int64_t slot_second2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_2 = HT_2.find(KEY_2[ITEM]);
if (SLOT_2 == HT_2.end()) {selection_flags[ITEM] = 0; continue;}
slot_second2[ITEM] = SLOT_2->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_lineitem__l_suppkey[ITEM];
KEY_4[ITEM] <<= 32;
KEY_4[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
//Create aggregation hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_4.insert(cuco::pair{KEY_4[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_FIND>
__global__ void main_5(uint64_t* BUF_2, HASHTABLE_PROBE HT_2, HASHTABLE_FIND HT_4, DBI32Type* KEY_4lineitem__l_partkey, DBI32Type* KEY_4lineitem__l_suppkey, DBDecimalType* aggr0__tmp_attr0, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI32Type* moved_aggr__ps_suppkey, DBI32Type* moved_aggr_u_1__ps_availqty, DBI32Type* moved_aggr_u_2__ps_partkey, DBI32Type* partsupp__ps_availqty, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBDateType reg_lineitem__l_shipdate[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_shipdate[ITEM] = lineitem__l_shipdate[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate[ITEM], 9131, Predicate::lt);
}
uint64_t KEY_2[ITEMS_PER_THREAD];
DBI32Type reg_lineitem__l_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_partkey[ITEM] = lineitem__l_partkey[ITEM*TB + tid];
}
DBI32Type reg_lineitem__l_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_suppkey[ITEM] = lineitem__l_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_2[ITEM] = 0;
KEY_2[ITEM] |= reg_lineitem__l_partkey[ITEM];
KEY_2[ITEM] <<= 32;
KEY_2[ITEM] |= reg_lineitem__l_suppkey[ITEM];
}
int64_t slot_second2[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_2 = HT_2.find(KEY_2[ITEM]);
if (SLOT_2 == HT_2.end()) {selection_flags[ITEM] = 0; continue;}
slot_second2[ITEM] = SLOT_2->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_4[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_4[ITEM] = 0;
KEY_4[ITEM] |= reg_lineitem__l_suppkey[ITEM];
KEY_4[ITEM] <<= 32;
KEY_4[ITEM] |= reg_lineitem__l_partkey[ITEM];
}
//Aggregate in hashtable
DBDecimalType reg_lineitem__l_quantity[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
reg_lineitem__l_quantity[ITEM] = lineitem__l_quantity[ITEM*TB + tid];
}
DBI32Type reg_partsupp__ps_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_partsupp__ps_suppkey[ITEM] = partsupp__ps_suppkey[BUF_2[slot_second2[ITEM] * 1 + 0]];
}
DBI32Type reg_partsupp__ps_availqty[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_partsupp__ps_availqty[ITEM] = partsupp__ps_availqty[BUF_2[slot_second2[ITEM] * 1 + 0]];
}
DBI32Type reg_partsupp__ps_partkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_partsupp__ps_partkey[ITEM] = partsupp__ps_partkey[BUF_2[slot_second2[ITEM] * 1 + 0]];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < lineitem_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_4 = HT_4.find(KEY_4[ITEM])->second;
aggregate_sum(&aggr0__tmp_attr0[buf_idx_4], reg_lineitem__l_quantity[ITEM]);
aggregate_any(&moved_aggr__ps_suppkey[buf_idx_4], reg_partsupp__ps_suppkey[ITEM]);
aggregate_any(&moved_aggr_u_1__ps_availqty[buf_idx_4], reg_partsupp__ps_availqty[ITEM]);
aggregate_any(&moved_aggr_u_2__ps_partkey[buf_idx_4], reg_partsupp__ps_partkey[ITEM]);
KEY_4lineitem__l_suppkey[buf_idx_4] = reg_lineitem__l_suppkey[ITEM];
KEY_4lineitem__l_partkey[buf_idx_4] = reg_lineitem__l_partkey[ITEM];
}
}
__global__ void count_7(uint64_t* COUNT6, DBStringType* nation__n_name, size_t nation_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_nation__n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_nation__n_name[ITEM] = nation__n_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_nation__n_name[ITEM], "CANADA", Predicate::eq);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT6, 1);
}
}
template<typename HASHTABLE_INSERT>
__global__ void main_7(uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_INSERT HT_6, DBStringType* nation__n_name, DBI32Type* nation__n_nationkey, size_t nation_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
DBStringType reg_nation__n_name[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_nation__n_name[ITEM] = nation__n_name[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= evaluatePredicate(reg_nation__n_name[ITEM], "CANADA", Predicate::eq);
}
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_nation__n_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
reg_nation__n_nationkey[ITEM] = nation__n_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_nation__n_nationkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < nation_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto buf_idx_6 = atomicAdd((int*)BUF_IDX_6, 1);
HT_6.insert(cuco::pair{KEY_6[ITEM], buf_idx_6});
BUF_6[(buf_idx_6) * 1 + 0] = ITEM*TB + tid;
}
}
__global__ void count_9(size_t COUNT4, uint64_t* COUNT8, DBDecimalType* aggr0__tmp_attr0, DBI32Type* moved_aggr__ps_suppkey, DBI32Type* moved_aggr_u_1__ps_availqty, DBI32Type* moved_aggr_u_2__ps_partkey) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
DBI32Type reg_partsupp__ps_availqty[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_partsupp__ps_availqty[ITEM] = moved_aggr_u_1__ps_availqty[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (0.5) * (reg_aggr0__tmp_attr0[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((true) && (evaluatePredicate(((DBDecimalType)reg_partsupp__ps_availqty[ITEM]), reg_map0__tmp_attr1[ITEM], Predicate::gt))) && (true);
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT8, 1);
}
}
template<typename HASHTABLE_INSERT_SJ>
__global__ void main_9(size_t COUNT4, HASHTABLE_INSERT_SJ HT_8, DBDecimalType* aggr0__tmp_attr0, DBI32Type* moved_aggr__ps_suppkey, DBI32Type* moved_aggr_u_1__ps_availqty, DBI32Type* moved_aggr_u_2__ps_partkey) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= !(false);
}
DBI32Type reg_partsupp__ps_availqty[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_partsupp__ps_availqty[ITEM] = moved_aggr_u_1__ps_availqty[ITEM*TB + tid];
}
DBDecimalType reg_aggr0__tmp_attr0[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_aggr0__tmp_attr0[ITEM] = aggr0__tmp_attr0[ITEM*TB + tid];
}
DBDecimalType reg_map0__tmp_attr1[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
if (!selection_flags[ITEM]) continue;
reg_map0__tmp_attr1[ITEM] = (0.5) * (reg_aggr0__tmp_attr0[ITEM]);
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= ((true) && (evaluatePredicate(((DBDecimalType)reg_partsupp__ps_availqty[ITEM]), reg_map0__tmp_attr1[ITEM], Predicate::gt))) && (true);
}
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_partsupp__ps_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
reg_partsupp__ps_suppkey[ITEM] = moved_aggr__ps_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_partsupp__ps_suppkey[ITEM];
}
// Insert hash table kernel;
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < COUNT4); ++ITEM) {
if (!selection_flags[ITEM]) continue;
HT_8.insert(cuco::pair{KEY_8[ITEM], 1});
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_PROBE_SJ>
__global__ void count_11(uint64_t* BUF_6, uint64_t* COUNT10, HASHTABLE_PROBE HT_6, HASHTABLE_PROBE_SJ HT_8, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_nationkey[ITEM] = supplier__s_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_supplier__s_nationkey[ITEM];
}
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
//Probe Hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_8 = HT_8.find(KEY_8[ITEM]);
if (SLOT_8 == HT_8.end()) {selection_flags[ITEM] = 0;}
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
//Materialize count
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
atomicAdd((int*)COUNT10, 1);
}
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_PROBE_SJ>
__global__ void main_11(uint64_t* BUF_6, HASHTABLE_PROBE HT_6, HASHTABLE_PROBE_SJ HT_8, DBI16Type* MAT10supplier__s_address_encoded, DBI16Type* MAT10supplier__s_name_encoded, uint64_t* MAT_IDX10, DBI16Type* supplier__s_address_encoded, DBI16Type* supplier__s_name_encoded, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
int64_t start, stop, cycles_per_warp;
size_t tile_offset = blockIdx.x * TILE_SIZE;
size_t tid = tile_offset + threadIdx.x;
int selection_flags[ITEMS_PER_THREAD];
for (int i=0; i<ITEMS_PER_THREAD; i++) selection_flags[i] = 1;
uint64_t KEY_6[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_nationkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_nationkey[ITEM] = supplier__s_nationkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_6[ITEM] = 0;
KEY_6[ITEM] |= reg_supplier__s_nationkey[ITEM];
}
int64_t slot_second6[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_6 = HT_6.find(KEY_6[ITEM]);
if (SLOT_6 == HT_6.end()) {selection_flags[ITEM] = 0; continue;}
slot_second6[ITEM] = SLOT_6->second;
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
uint64_t KEY_8[ITEMS_PER_THREAD];
DBI32Type reg_supplier__s_suppkey[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_suppkey[ITEM] = supplier__s_suppkey[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
KEY_8[ITEM] = 0;
KEY_8[ITEM] |= reg_supplier__s_suppkey[ITEM];
}
//Probe Hash table
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto SLOT_8 = HT_8.find(KEY_8[ITEM]);
if (SLOT_8 == HT_8.end()) {selection_flags[ITEM] = 0;}
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
selection_flags[ITEM] &= true;
}
//Materialize buffers
DBI16Type reg_supplier__s_name_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_name_encoded[ITEM] = supplier__s_name_encoded[ITEM*TB + tid];
}
DBI16Type reg_supplier__s_address_encoded[ITEMS_PER_THREAD];
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
reg_supplier__s_address_encoded[ITEM] = supplier__s_address_encoded[ITEM*TB + tid];
}
#pragma unroll
for (int ITEM = 0; ITEM < ITEMS_PER_THREAD && (ITEM*TB + tid < supplier_size); ++ITEM) {
if (!selection_flags[ITEM]) continue;
auto mat_idx10 = atomicAdd((int*)MAT_IDX10, 1);
MAT10supplier__s_name_encoded[mat_idx10] = reg_supplier__s_name_encoded[ITEM];
MAT10supplier__s_address_encoded[mat_idx10] = reg_supplier__s_address_encoded[ITEM];
}
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)part_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT0, d_part__p_name, part_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)part_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_HT_0.ref(cuco::insert), d_part__p_name, d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT2;
hipMalloc(&d_COUNT2, sizeof(uint64_t));
hipMemset(d_COUNT2, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)partsupp_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT2, d_HT_0.ref(cuco::find), d_partsupp__ps_partkey, partsupp_size);
uint64_t COUNT2;
hipMemcpy(&COUNT2, d_COUNT2, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 1);
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)partsupp_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_2, d_BUF_IDX_2, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::insert), d_partsupp__ps_partkey, d_partsupp__ps_suppkey, partsupp_size);
//Create aggregation hash table
auto d_HT_4 = cuco::static_map{ (int)9138501*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_2, d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::insert), d_lineitem__l_partkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size);
size_t COUNT4 = d_HT_4.size();
thrust::device_vector<int64_t> keys_4(COUNT4), vals_4(COUNT4);
d_HT_4.retrieve_all(keys_4.begin(), vals_4.begin());
d_HT_4.clear();
int64_t* raw_keys4 = thrust::raw_pointer_cast(keys_4.data());
insertKeys<<<std::ceil((float)COUNT4/128.), 128>>>(raw_keys4, d_HT_4.ref(cuco::insert), COUNT4);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT4);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT4);
DBI32Type* d_moved_aggr__ps_suppkey;
hipMalloc(&d_moved_aggr__ps_suppkey, sizeof(DBI32Type) * COUNT4);
hipMemset(d_moved_aggr__ps_suppkey, 0, sizeof(DBI32Type) * COUNT4);
DBI32Type* d_moved_aggr_u_1__ps_availqty;
hipMalloc(&d_moved_aggr_u_1__ps_availqty, sizeof(DBI32Type) * COUNT4);
hipMemset(d_moved_aggr_u_1__ps_availqty, 0, sizeof(DBI32Type) * COUNT4);
DBI32Type* d_moved_aggr_u_2__ps_partkey;
hipMalloc(&d_moved_aggr_u_2__ps_partkey, sizeof(DBI32Type) * COUNT4);
hipMemset(d_moved_aggr_u_2__ps_partkey, 0, sizeof(DBI32Type) * COUNT4);
DBI32Type* d_KEY_4lineitem__l_suppkey;
hipMalloc(&d_KEY_4lineitem__l_suppkey, sizeof(DBI32Type) * COUNT4);
hipMemset(d_KEY_4lineitem__l_suppkey, 0, sizeof(DBI32Type) * COUNT4);
DBI32Type* d_KEY_4lineitem__l_partkey;
hipMalloc(&d_KEY_4lineitem__l_partkey, sizeof(DBI32Type) * COUNT4);
hipMemset(d_KEY_4lineitem__l_partkey, 0, sizeof(DBI32Type) * COUNT4);
main_5<<<std::ceil((float)lineitem_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_2, d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::find), d_KEY_4lineitem__l_partkey, d_KEY_4lineitem__l_suppkey, d_aggr0__tmp_attr0, d_lineitem__l_partkey, d_lineitem__l_quantity, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_moved_aggr__ps_suppkey, d_moved_aggr_u_1__ps_availqty, d_moved_aggr_u_2__ps_partkey, d_partsupp__ps_availqty, d_partsupp__ps_partkey, d_partsupp__ps_suppkey);
//Materialize count
uint64_t* d_COUNT6;
hipMalloc(&d_COUNT6, sizeof(uint64_t));
hipMemset(d_COUNT6, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_COUNT6, d_nation__n_name, nation_size);
uint64_t COUNT6;
hipMemcpy(&COUNT6, d_COUNT6, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 1);
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)nation_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_6, d_BUF_IDX_6, d_HT_6.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
//Materialize count
uint64_t* d_COUNT8;
hipMalloc(&d_COUNT8, sizeof(uint64_t));
hipMemset(d_COUNT8, 0, sizeof(uint64_t));
count_9<<<std::ceil((float)COUNT4/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT4, d_COUNT8, d_aggr0__tmp_attr0, d_moved_aggr__ps_suppkey, d_moved_aggr_u_1__ps_availqty, d_moved_aggr_u_2__ps_partkey);
uint64_t COUNT8;
hipMemcpy(&COUNT8, d_COUNT8, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
auto d_HT_8 = cuco::static_map{ (int)COUNT8*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_9<<<std::ceil((float)COUNT4/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(COUNT4, d_HT_8.ref(cuco::insert), d_aggr0__tmp_attr0, d_moved_aggr__ps_suppkey, d_moved_aggr_u_1__ps_availqty, d_moved_aggr_u_2__ps_partkey);
//Materialize count
uint64_t* d_COUNT10;
hipMalloc(&d_COUNT10, sizeof(uint64_t));
hipMemset(d_COUNT10, 0, sizeof(uint64_t));
count_11<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_6, d_COUNT10, d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::find), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
uint64_t COUNT10;
hipMemcpy(&COUNT10, d_COUNT10, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX10;
hipMalloc(&d_MAT_IDX10, sizeof(uint64_t));
hipMemset(d_MAT_IDX10, 0, sizeof(uint64_t));
auto MAT10supplier__s_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT10);
DBI16Type* d_MAT10supplier__s_name_encoded;
hipMalloc(&d_MAT10supplier__s_name_encoded, sizeof(DBI16Type) * COUNT10);
auto MAT10supplier__s_address_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT10);
DBI16Type* d_MAT10supplier__s_address_encoded;
hipMalloc(&d_MAT10supplier__s_address_encoded, sizeof(DBI16Type) * COUNT10);
main_11<<<std::ceil((float)supplier_size/(float)TILE_SIZE), TILE_SIZE/ITEMS_PER_THREAD>>>(d_BUF_6, d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::find), d_MAT10supplier__s_address_encoded, d_MAT10supplier__s_name_encoded, d_MAT_IDX10, d_supplier__s_address_encoded, d_supplier__s_name_encoded, d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
hipMemcpy(MAT10supplier__s_name_encoded, d_MAT10supplier__s_name_encoded, sizeof(DBI16Type) * COUNT10, hipMemcpyDeviceToHost);
hipMemcpy(MAT10supplier__s_address_encoded, d_MAT10supplier__s_address_encoded, sizeof(DBI16Type) * COUNT10, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < COUNT10; i++) { std::cout << "" << supplier__s_name_map[MAT10supplier__s_name_encoded[i]];
std::cout << "|" << supplier__s_address_map[MAT10supplier__s_address_encoded[i]];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
      size_t aux_mem = usedGpuMem() - used_mem;
      std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_COUNT0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_COUNT2);
hipFree(d_KEY_4lineitem__l_partkey);
hipFree(d_KEY_4lineitem__l_suppkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_moved_aggr__ps_suppkey);
hipFree(d_moved_aggr_u_1__ps_availqty);
hipFree(d_moved_aggr_u_2__ps_partkey);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_COUNT6);
hipFree(d_COUNT8);
hipFree(d_COUNT10);
hipFree(d_MAT10supplier__s_address_encoded);
hipFree(d_MAT10supplier__s_name_encoded);
hipFree(d_MAT_IDX10);
free(MAT10supplier__s_address_encoded);
free(MAT10supplier__s_name_encoded);
}