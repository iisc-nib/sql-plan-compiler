#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
template<typename HASHTABLE_INSERT>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT HT_0, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
uint64_t KEY_0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_0 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0, buf_idx_0});
BUF_0[buf_idx_0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_FIND>
__global__ void main_3(uint64_t* BUF_0, HASHTABLE_PROBE_PK HT_0, HASHTABLE_FIND HT_2, int* SLOT_COUNT_2, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_partkey, DBDateType* lineitem__l_shipdate, size_t lineitem_size, DBStringType* part__p_type) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9374, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9404, Predicate::lt))) return;
uint64_t KEY_0 = 0;
auto reg_lineitem__l_partkey = lineitem__l_partkey[tid];

KEY_0 |= reg_lineitem__l_partkey;
//Probe Hash table
auto SLOT_0 = HT_0.find(KEY_0);
if (SLOT_0 == HT_0.end()) return;
if (!(true)) return;
auto reg_part__p_type = part__p_type[BUF_0[SLOT_0->second * 1 + 0]];
uint64_t KEY_2 = 0;
//Aggregate in hashtable
auto buf_idx_2 = get_aggregation_slot(KEY_2, HT_2, SLOT_COUNT_2);
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr3 = (reg_lineitem__l_extendedprice) * ((1.0) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_2], reg_map0__tmp_attr3);
auto reg_map0__tmp_attr1 = ((Like(reg_part__p_type, "PROMO", "", nullptr, nullptr, 0))) ? ((reg_lineitem__l_extendedprice) * ((1.0) - (reg_lineitem__l_discount))) : (0.0);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_2], reg_map0__tmp_attr1);
}
__global__ void main_5(size_t COUNT2, DBDecimalType* MAT4map1__tmp_attr4, uint64_t* MAT_IDX4, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr0__tmp_attr2) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT2) return;
//Materialize buffers
auto mat_idx4 = atomicAdd((int*)MAT_IDX4, 1);
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
auto reg_map1__tmp_attr4 = ((100.00) * (reg_aggr0__tmp_attr0)) / (reg_aggr0__tmp_attr2);
MAT4map1__tmp_attr4[mat_idx4] = reg_map1__tmp_attr4;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
size_t COUNT0 = part_size;
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)part_size/128.), 128>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_part__p_partkey, part_size);
hipFree(d_BUF_IDX_0);
size_t COUNT2 = 1;
auto d_HT_2 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},         cuco::empty_value{(int64_t)-1},         thrust::equal_to<int64_t>{},         cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
int* d_SLOT_COUNT_2;
hipMalloc(&d_SLOT_COUNT_2, sizeof(int));
hipMemset(d_SLOT_COUNT_2, 0, sizeof(int));
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT2);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT2);
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT2);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT2);
main_3<<<std::ceil((float)lineitem_size/128.), 128>>>(d_BUF_0, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::insert_and_find), d_SLOT_COUNT_2, d_aggr0__tmp_attr0, d_aggr0__tmp_attr2, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_partkey, d_lineitem__l_shipdate, lineitem_size, d_part__p_type);
COUNT2 = d_HT_2.size();
size_t COUNT4 = COUNT2;
//Materialize buffers
uint64_t* d_MAT_IDX4;
hipMalloc(&d_MAT_IDX4, sizeof(uint64_t));
hipMemset(d_MAT_IDX4, 0, sizeof(uint64_t));
auto MAT4map1__tmp_attr4 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT4);
DBDecimalType* d_MAT4map1__tmp_attr4;
hipMalloc(&d_MAT4map1__tmp_attr4, sizeof(DBDecimalType) * COUNT4);
main_5<<<std::ceil((float)COUNT2/128.), 128>>>(COUNT2, d_MAT4map1__tmp_attr4, d_MAT_IDX4, d_aggr0__tmp_attr0, d_aggr0__tmp_attr2);
uint64_t MATCOUNT_4 = 0;
hipMemcpy(&MATCOUNT_4, d_MAT_IDX4, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipMemcpy(MAT4map1__tmp_attr4, d_MAT4map1__tmp_attr4, sizeof(DBDecimalType) * COUNT4, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < MATCOUNT_4; i++) { std::cout << "" << MAT4map1__tmp_attr4[i];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_MAT4map1__tmp_attr4);
hipFree(d_MAT_IDX4);
free(MAT4map1__tmp_attr4);
}