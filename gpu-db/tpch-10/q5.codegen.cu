#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
__global__ void count_1(uint64_t* COUNT0, DBStringType* region__r_name, size_t region_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= region_size) return;
auto reg_region__r_name = region__r_name[tid];
if (!(evaluatePredicate(reg_region__r_name, "ASIA", Predicate::eq))) return;
//Materialize count
atomicAdd((int*)COUNT0, 1);
}
template<typename HASHTABLE_INSERT_PK>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT_PK HT_0, DBStringType* region__r_name, DBI32Type* region__r_regionkey, size_t region_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= region_size) return;
auto reg_region__r_name = region__r_name[tid];
if (!(evaluatePredicate(reg_region__r_name, "ASIA", Predicate::eq))) return;
uint64_t KEY_0 = 0;
auto reg_region__r_regionkey = region__r_regionkey[tid];

KEY_0 |= reg_region__r_regionkey;
// Insert hash table kernel;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0, buf_idx_0});
BUF_0[buf_idx_0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE_PK>
__global__ void count_3(uint64_t* BUF_0, uint64_t* COUNT2, HASHTABLE_PROBE_PK HT_0, DBI32Type* nation__n_regionkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_0 = 0;
auto reg_nation__n_regionkey = nation__n_regionkey[tid];

KEY_0 |= reg_nation__n_regionkey;
//Probe Hash table
auto SLOT_0 = HT_0.find(KEY_0);
if (SLOT_0 == HT_0.end()) return;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT2, 1);
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_INSERT_PK>
__global__ void main_3(uint64_t* BUF_0, uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_PROBE_PK HT_0, HASHTABLE_INSERT_PK HT_2, DBI32Type* nation__n_nationkey, DBI32Type* nation__n_regionkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_0 = 0;
auto reg_nation__n_regionkey = nation__n_regionkey[tid];

KEY_0 |= reg_nation__n_regionkey;
//Probe Hash table
auto SLOT_0 = HT_0.find(KEY_0);
if (SLOT_0 == HT_0.end()) return;
if (!(true)) return;
uint64_t KEY_2 = 0;
auto reg_nation__n_nationkey = nation__n_nationkey[tid];

KEY_2 |= reg_nation__n_nationkey;
// Insert hash table kernel;
auto buf_idx_2 = atomicAdd((int*)BUF_IDX_2, 1);
HT_2.insert(cuco::pair{KEY_2, buf_idx_2});
BUF_2[buf_idx_2 * 2 + 0] = BUF_0[SLOT_0->second * 1 + 0];
BUF_2[buf_idx_2 * 2 + 1] = tid;
}
template<typename HASHTABLE_PROBE_PK>
__global__ void count_5(uint64_t* BUF_2, uint64_t* COUNT4, HASHTABLE_PROBE_PK HT_2, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_2 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_2 |= reg_customer__c_nationkey;
//Probe Hash table
auto SLOT_2 = HT_2.find(KEY_2);
if (SLOT_2 == HT_2.end()) return;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT4, 1);
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_INSERT_PK>
__global__ void main_5(uint64_t* BUF_2, uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_PROBE_PK HT_2, HASHTABLE_INSERT_PK HT_4, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_2 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_2 |= reg_customer__c_nationkey;
//Probe Hash table
auto SLOT_2 = HT_2.find(KEY_2);
if (SLOT_2 == HT_2.end()) return;
if (!(true)) return;
uint64_t KEY_4 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_4 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_4 = atomicAdd((int*)BUF_IDX_4, 1);
HT_4.insert(cuco::pair{KEY_4, buf_idx_4});
BUF_4[buf_idx_4 * 3 + 0] = tid;
BUF_4[buf_idx_4 * 3 + 1] = BUF_2[SLOT_2->second * 2 + 0];
BUF_4[buf_idx_4 * 3 + 2] = BUF_2[SLOT_2->second * 2 + 1];
}
template<typename HASHTABLE_PROBE_PK>
__global__ void count_7(uint64_t* BUF_4, uint64_t* COUNT6, HASHTABLE_PROBE_PK HT_4, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 9131, Predicate::lt))) return;
uint64_t KEY_4 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_4 |= reg_orders__o_custkey;
//Probe Hash table
auto SLOT_4 = HT_4.find(KEY_4);
if (SLOT_4 == HT_4.end()) return;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT6, 1);
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_INSERT_PK>
__global__ void main_7(uint64_t* BUF_4, uint64_t* BUF_6, uint64_t* BUF_IDX_6, HASHTABLE_PROBE_PK HT_4, HASHTABLE_INSERT_PK HT_6, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 9131, Predicate::lt))) return;
uint64_t KEY_4 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_4 |= reg_orders__o_custkey;
//Probe Hash table
auto SLOT_4 = HT_4.find(KEY_4);
if (SLOT_4 == HT_4.end()) return;
if (!(true)) return;
uint64_t KEY_6 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_6 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_6 = atomicAdd((int*)BUF_IDX_6, 1);
HT_6.insert(cuco::pair{KEY_6, buf_idx_6});
BUF_6[buf_idx_6 * 4 + 0] = BUF_4[SLOT_4->second * 3 + 0];
BUF_6[buf_idx_6 * 4 + 1] = tid;
BUF_6[buf_idx_6 * 4 + 2] = BUF_4[SLOT_4->second * 3 + 1];
BUF_6[buf_idx_6 * 4 + 3] = BUF_4[SLOT_4->second * 3 + 2];
}
__global__ void count_9(uint64_t* COUNT8, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
//Materialize count
atomicAdd((int*)COUNT8, 1);
}
template<typename HASHTABLE_INSERT_PK>
__global__ void main_9(uint64_t* BUF_8, uint64_t* BUF_IDX_8, HASHTABLE_INSERT_PK HT_8, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_8 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_8 |= reg_supplier__s_suppkey;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];
KEY_8 <<= 32;
KEY_8 |= reg_supplier__s_nationkey;
// Insert hash table kernel;
auto buf_idx_8 = atomicAdd((int*)BUF_IDX_8, 1);
HT_8.insert(cuco::pair{KEY_8, buf_idx_8});
BUF_8[buf_idx_8 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE_PK>
__global__ void count_11(uint64_t* BUF_6, uint64_t* BUF_8, HASHTABLE_INSERT HT_10, HASHTABLE_PROBE_PK HT_6, HASHTABLE_PROBE_PK HT_8, DBI32Type* customer__c_nationkey, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_6 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_6 |= reg_lineitem__l_orderkey;
//Probe Hash table
auto SLOT_6 = HT_6.find(KEY_6);
if (SLOT_6 == HT_6.end()) return;
if (!(true)) return;
uint64_t KEY_8 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_8 |= reg_lineitem__l_suppkey;
auto reg_customer__c_nationkey = customer__c_nationkey[BUF_6[SLOT_6->second * 4 + 0]];
KEY_8 <<= 32;
KEY_8 |= reg_customer__c_nationkey;
//Probe Hash table
auto SLOT_8 = HT_8.find(KEY_8);
if (SLOT_8 == HT_8.end()) return;
if (!(true)) return;
uint64_t KEY_10 = 0;
auto reg_nation__n_name_encoded = nation__n_name_encoded[BUF_6[SLOT_6->second * 4 + 3]];

KEY_10 |= reg_nation__n_name_encoded;
//Create aggregation hash table
HT_10.insert(cuco::pair{KEY_10, 1});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE_PK>
__global__ void main_11(uint64_t* BUF_6, uint64_t* BUF_8, HASHTABLE_FIND HT_10, HASHTABLE_PROBE_PK HT_6, HASHTABLE_PROBE_PK HT_8, DBI16Type* KEY_10nation__n_name_encoded, DBDecimalType* aggr0__tmp_attr0, DBI32Type* customer__c_nationkey, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBI16Type* nation__n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_6 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_6 |= reg_lineitem__l_orderkey;
//Probe Hash table
auto SLOT_6 = HT_6.find(KEY_6);
if (SLOT_6 == HT_6.end()) return;
if (!(true)) return;
uint64_t KEY_8 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_8 |= reg_lineitem__l_suppkey;
auto reg_customer__c_nationkey = customer__c_nationkey[BUF_6[SLOT_6->second * 4 + 0]];
KEY_8 <<= 32;
KEY_8 |= reg_customer__c_nationkey;
//Probe Hash table
auto SLOT_8 = HT_8.find(KEY_8);
if (SLOT_8 == HT_8.end()) return;
if (!(true)) return;
uint64_t KEY_10 = 0;
auto reg_nation__n_name_encoded = nation__n_name_encoded[BUF_6[SLOT_6->second * 4 + 3]];

KEY_10 |= reg_nation__n_name_encoded;
//Aggregate in hashtable
auto buf_idx_10 = HT_10.find(KEY_10)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1.0) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_10], reg_map0__tmp_attr1);
KEY_10nation__n_name_encoded[buf_idx_10] = reg_nation__n_name_encoded;
}
__global__ void count_13(size_t COUNT10, uint64_t* COUNT12) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT10) return;
//Materialize count
atomicAdd((int*)COUNT12, 1);
}
__global__ void main_13(size_t COUNT10, DBDecimalType* MAT12aggr0__tmp_attr0, DBI16Type* MAT12nation__n_name_encoded, uint64_t* MAT_IDX12, DBDecimalType* aggr0__tmp_attr0, DBI16Type* nation__n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT10) return;
//Materialize buffers
auto mat_idx12 = atomicAdd((int*)MAT_IDX12, 1);
auto reg_nation__n_name_encoded = nation__n_name_encoded[tid];
MAT12nation__n_name_encoded[mat_idx12] = reg_nation__n_name_encoded;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT12aggr0__tmp_attr0[mat_idx12] = reg_aggr0__tmp_attr0;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)region_size/128.), 128>>>(d_COUNT0, d_region__r_name, region_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)region_size/128.), 128>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_region__r_name, d_region__r_regionkey, region_size);
//Materialize count
uint64_t* d_COUNT2;
hipMalloc(&d_COUNT2, sizeof(uint64_t));
hipMemset(d_COUNT2, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)nation_size/128.), 128>>>(d_BUF_0, d_COUNT2, d_HT_0.ref(cuco::find), d_nation__n_regionkey, nation_size);
uint64_t COUNT2;
hipMemcpy(&COUNT2, d_COUNT2, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 2);
auto d_HT_2 = cuco::static_map{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)nation_size/128.), 128>>>(d_BUF_0, d_BUF_2, d_BUF_IDX_2, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::insert), d_nation__n_nationkey, d_nation__n_regionkey, nation_size);
//Materialize count
uint64_t* d_COUNT4;
hipMalloc(&d_COUNT4, sizeof(uint64_t));
hipMemset(d_COUNT4, 0, sizeof(uint64_t));
count_5<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_2, d_COUNT4, d_HT_2.ref(cuco::find), d_customer__c_nationkey, customer_size);
uint64_t COUNT4;
hipMemcpy(&COUNT4, d_COUNT4, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 3);
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_2, d_BUF_4, d_BUF_IDX_4, d_HT_2.ref(cuco::find), d_HT_4.ref(cuco::insert), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
//Materialize count
uint64_t* d_COUNT6;
hipMalloc(&d_COUNT6, sizeof(uint64_t));
hipMemset(d_COUNT6, 0, sizeof(uint64_t));
count_7<<<std::ceil((float)orders_size/128.), 128>>>(d_BUF_4, d_COUNT6, d_HT_4.ref(cuco::find), d_orders__o_custkey, d_orders__o_orderdate, orders_size);
uint64_t COUNT6;
hipMemcpy(&COUNT6, d_COUNT6, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_6;
hipMalloc(&d_BUF_IDX_6, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6, 0, sizeof(uint64_t));
uint64_t* d_BUF_6;
hipMalloc(&d_BUF_6, sizeof(uint64_t) * COUNT6 * 4);
auto d_HT_6 = cuco::static_map{ (int)COUNT6*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_7<<<std::ceil((float)orders_size/128.), 128>>>(d_BUF_4, d_BUF_6, d_BUF_IDX_6, d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
//Materialize count
uint64_t* d_COUNT8;
hipMalloc(&d_COUNT8, sizeof(uint64_t));
hipMemset(d_COUNT8, 0, sizeof(uint64_t));
count_9<<<std::ceil((float)supplier_size/128.), 128>>>(d_COUNT8, supplier_size);
uint64_t COUNT8;
hipMemcpy(&COUNT8, d_COUNT8, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_8;
hipMalloc(&d_BUF_IDX_8, sizeof(uint64_t));
hipMemset(d_BUF_IDX_8, 0, sizeof(uint64_t));
uint64_t* d_BUF_8;
hipMalloc(&d_BUF_8, sizeof(uint64_t) * COUNT8 * 1);
auto d_HT_8 = cuco::static_map{ (int)COUNT8*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_9<<<std::ceil((float)supplier_size/128.), 128>>>(d_BUF_8, d_BUF_IDX_8, d_HT_8.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
//Create aggregation hash table
auto d_HT_10 = cuco::static_map{ (int)234321*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_11<<<std::ceil((float)lineitem_size/128.), 128>>>(d_BUF_6, d_BUF_8, d_HT_10.ref(cuco::insert), d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::find), d_customer__c_nationkey, d_lineitem__l_orderkey, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded);
size_t COUNT10 = d_HT_10.size();
thrust::device_vector<int64_t> keys_10(COUNT10), vals_10(COUNT10);
d_HT_10.retrieve_all(keys_10.begin(), vals_10.begin());
d_HT_10.clear();
int64_t* raw_keys10 = thrust::raw_pointer_cast(keys_10.data());
insertKeys<<<std::ceil((float)COUNT10/128.), 128>>>(raw_keys10, d_HT_10.ref(cuco::insert), COUNT10);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT10);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT10);
DBI16Type* d_KEY_10nation__n_name_encoded;
hipMalloc(&d_KEY_10nation__n_name_encoded, sizeof(DBI16Type) * COUNT10);
hipMemset(d_KEY_10nation__n_name_encoded, 0, sizeof(DBI16Type) * COUNT10);
main_11<<<std::ceil((float)lineitem_size/128.), 128>>>(d_BUF_6, d_BUF_8, d_HT_10.ref(cuco::find), d_HT_6.ref(cuco::find), d_HT_8.ref(cuco::find), d_KEY_10nation__n_name_encoded, d_aggr0__tmp_attr0, d_customer__c_nationkey, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name_encoded);
//Materialize count
uint64_t* d_COUNT12;
hipMalloc(&d_COUNT12, sizeof(uint64_t));
hipMemset(d_COUNT12, 0, sizeof(uint64_t));
count_13<<<std::ceil((float)COUNT10/128.), 128>>>(COUNT10, d_COUNT12);
uint64_t COUNT12;
hipMemcpy(&COUNT12, d_COUNT12, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX12;
hipMalloc(&d_MAT_IDX12, sizeof(uint64_t));
hipMemset(d_MAT_IDX12, 0, sizeof(uint64_t));
auto MAT12nation__n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT12);
DBI16Type* d_MAT12nation__n_name_encoded;
hipMalloc(&d_MAT12nation__n_name_encoded, sizeof(DBI16Type) * COUNT12);
auto MAT12aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT12);
DBDecimalType* d_MAT12aggr0__tmp_attr0;
hipMalloc(&d_MAT12aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT12);
main_13<<<std::ceil((float)COUNT10/128.), 128>>>(COUNT10, d_MAT12aggr0__tmp_attr0, d_MAT12nation__n_name_encoded, d_MAT_IDX12, d_aggr0__tmp_attr0, d_KEY_10nation__n_name_encoded);
hipMemcpy(MAT12nation__n_name_encoded, d_MAT12nation__n_name_encoded, sizeof(DBI16Type) * COUNT12, hipMemcpyDeviceToHost);
hipMemcpy(MAT12aggr0__tmp_attr0, d_MAT12aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT12, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < COUNT12; i++) { std::cout << "" << nation__n_name_map[MAT12nation__n_name_encoded[i]];
std::cout << "|" << MAT12aggr0__tmp_attr0[i];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_COUNT0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_COUNT2);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_COUNT4);
hipFree(d_BUF_6);
hipFree(d_BUF_IDX_6);
hipFree(d_COUNT6);
hipFree(d_BUF_8);
hipFree(d_BUF_IDX_8);
hipFree(d_COUNT8);
hipFree(d_KEY_10nation__n_name_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT12);
hipFree(d_MAT12aggr0__tmp_attr0);
hipFree(d_MAT12nation__n_name_encoded);
hipFree(d_MAT_IDX12);
free(MAT12aggr0__tmp_attr0);
free(MAT12nation__n_name_encoded);
}