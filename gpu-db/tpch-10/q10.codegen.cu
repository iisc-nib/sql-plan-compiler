#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
__global__ void count_1(uint64_t* COUNT0, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8674, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::lt))) return;
//Materialize count
atomicAdd((int*)COUNT0, 1);
}
template<typename HASHTABLE_INSERT_PK>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT_PK HT_0, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8674, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::lt))) return;
uint64_t KEY_0 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_0 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0, buf_idx_0});
BUF_0[buf_idx_0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE_PK>
__global__ void count_3(uint64_t* BUF_0, uint64_t* COUNT2, HASHTABLE_PROBE_PK HT_0, DBI32Type* lineitem__l_orderkey, DBCharType* lineitem__l_returnflag, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];
if (!(evaluatePredicate(reg_lineitem__l_returnflag, 'R', Predicate::eq))) return;
uint64_t KEY_0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_0 |= reg_lineitem__l_orderkey;
//Probe Hash table
auto SLOT_0 = HT_0.find(KEY_0);
if (SLOT_0 == HT_0.end()) return;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT2, 1);
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_INSERT>
__global__ void main_3(uint64_t* BUF_0, uint64_t* BUF_2, uint64_t* BUF_IDX_2, HASHTABLE_PROBE_PK HT_0, HASHTABLE_INSERT HT_2, DBI32Type* lineitem__l_orderkey, DBCharType* lineitem__l_returnflag, size_t lineitem_size, DBI32Type* orders__o_custkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];
if (!(evaluatePredicate(reg_lineitem__l_returnflag, 'R', Predicate::eq))) return;
uint64_t KEY_0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_0 |= reg_lineitem__l_orderkey;
//Probe Hash table
auto SLOT_0 = HT_0.find(KEY_0);
if (SLOT_0 == HT_0.end()) return;
if (!(true)) return;
uint64_t KEY_2 = 0;
auto reg_orders__o_custkey = orders__o_custkey[BUF_0[SLOT_0->second * 1 + 0]];

KEY_2 |= reg_orders__o_custkey;
// Insert hash table kernel;
auto buf_idx_2 = atomicAdd((int*)BUF_IDX_2, 1);
HT_2.insert(cuco::pair{KEY_2, buf_idx_2});
BUF_2[buf_idx_2 * 2 + 0] = BUF_0[SLOT_0->second * 1 + 0];
BUF_2[buf_idx_2 * 2 + 1] = tid;
}
__global__ void count_5(uint64_t* COUNT4, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
//Materialize count
atomicAdd((int*)COUNT4, 1);
}
template<typename HASHTABLE_INSERT_PK>
__global__ void main_5(uint64_t* BUF_4, uint64_t* BUF_IDX_4, HASHTABLE_INSERT_PK HT_4, DBI32Type* nation__n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_4 = 0;
auto reg_nation__n_nationkey = nation__n_nationkey[tid];

KEY_4 |= reg_nation__n_nationkey;
// Insert hash table kernel;
auto buf_idx_4 = atomicAdd((int*)BUF_IDX_4, 1);
HT_4.insert(cuco::pair{KEY_4, buf_idx_4});
BUF_4[buf_idx_4 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_PROBE_PK, typename HASHTABLE_INSERT>
__global__ void count_7(uint64_t* BUF_2, uint64_t* BUF_4, HASHTABLE_PROBE HT_2, HASHTABLE_PROBE_PK HT_4, HASHTABLE_INSERT HT_6, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_2 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_2 |= reg_customer__c_custkey;
//Probe Hash table
HT_2.for_each(KEY_2, [&] __device__ (auto const SLOT_2) {

auto const [slot_first2, slot_second2] = SLOT_2;
if (!(true)) return;
uint64_t KEY_4 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_4 |= reg_customer__c_nationkey;
//Probe Hash table
auto SLOT_4 = HT_4.find(KEY_4);
if (SLOT_4 == HT_4.end()) return;
if (!(true)) return;
uint64_t KEY_6 = 0;

KEY_6 |= reg_customer__c_custkey;
//Create aggregation hash table
HT_6.insert(cuco::pair{KEY_6, 1});
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_PROBE_PK, typename HASHTABLE_FIND>
__global__ void main_7(uint64_t* BUF_2, uint64_t* BUF_4, HASHTABLE_PROBE HT_2, HASHTABLE_PROBE_PK HT_4, HASHTABLE_FIND HT_6, DBI32Type* KEY_6customer__c_custkey, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr__c_acctbal, DBI16Type* aggr__n_name_encoded, DBDecimalType* customer__c_acctbal, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI16Type* nation__n_name_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_2 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_2 |= reg_customer__c_custkey;
//Probe Hash table
HT_2.for_each(KEY_2, [&] __device__ (auto const SLOT_2) {
auto const [slot_first2, slot_second2] = SLOT_2;
if (!(true)) return;
uint64_t KEY_4 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_4 |= reg_customer__c_nationkey;
//Probe Hash table
auto SLOT_4 = HT_4.find(KEY_4);
if (SLOT_4 == HT_4.end()) return;
if (!(true)) return;
uint64_t KEY_6 = 0;

KEY_6 |= reg_customer__c_custkey;
//Aggregate in hashtable
auto buf_idx_6 = HT_6.find(KEY_6)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[BUF_2[slot_second2 * 2 + 1]];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[BUF_2[slot_second2 * 2 + 1]];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1.0) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_6], reg_map0__tmp_attr1);
auto reg_nation__n_name_encoded = nation__n_name_encoded[BUF_4[SLOT_4->second * 1 + 0]];
aggregate_any(&aggr__n_name_encoded[buf_idx_6], reg_nation__n_name_encoded);
auto reg_customer__c_acctbal = customer__c_acctbal[tid];
aggregate_any(&aggr__c_acctbal[buf_idx_6], reg_customer__c_acctbal);
KEY_6customer__c_custkey[buf_idx_6] = reg_customer__c_custkey;
});
}
__global__ void count_9(size_t COUNT6, uint64_t* COUNT8) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT6) return;
//Materialize count
atomicAdd((int*)COUNT8, 1);
}
__global__ void main_9(size_t COUNT6, DBDecimalType* MAT8aggr0__tmp_attr0, DBDecimalType* MAT8aggr__c_acctbal, DBI16Type* MAT8aggr__n_name_encoded, DBI32Type* MAT8customer__c_custkey, uint64_t* MAT_IDX8, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr__c_acctbal, DBI16Type* aggr__n_name_encoded, DBI32Type* customer__c_custkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT6) return;
//Materialize buffers
auto mat_idx8 = atomicAdd((int*)MAT_IDX8, 1);
auto reg_customer__c_custkey = customer__c_custkey[tid];
MAT8customer__c_custkey[mat_idx8] = reg_customer__c_custkey;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT8aggr0__tmp_attr0[mat_idx8] = reg_aggr0__tmp_attr0;
auto reg_aggr__c_acctbal = aggr__c_acctbal[tid];
MAT8aggr__c_acctbal[mat_idx8] = reg_aggr__c_acctbal;
auto reg_aggr__n_name_encoded = aggr__n_name_encoded[tid];
MAT8aggr__n_name_encoded[mat_idx8] = reg_aggr__n_name_encoded;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)orders_size/128.), 128>>>(d_COUNT0, d_orders__o_orderdate, orders_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)orders_size/128.), 128>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
//Materialize count
uint64_t* d_COUNT2;
hipMalloc(&d_COUNT2, sizeof(uint64_t));
hipMemset(d_COUNT2, 0, sizeof(uint64_t));
count_3<<<std::ceil((float)lineitem_size/128.), 128>>>(d_BUF_0, d_COUNT2, d_HT_0.ref(cuco::find), d_lineitem__l_orderkey, d_lineitem__l_returnflag, lineitem_size);
uint64_t COUNT2;
hipMemcpy(&COUNT2, d_COUNT2, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_2;
hipMalloc(&d_BUF_IDX_2, sizeof(uint64_t));
hipMemset(d_BUF_IDX_2, 0, sizeof(uint64_t));
uint64_t* d_BUF_2;
hipMalloc(&d_BUF_2, sizeof(uint64_t) * COUNT2 * 2);
auto d_HT_2 = cuco::experimental::static_multimap{ (int)COUNT2*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_3<<<std::ceil((float)lineitem_size/128.), 128>>>(d_BUF_0, d_BUF_2, d_BUF_IDX_2, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::insert), d_lineitem__l_orderkey, d_lineitem__l_returnflag, lineitem_size, d_orders__o_custkey);
//Materialize count
uint64_t* d_COUNT4;
hipMalloc(&d_COUNT4, sizeof(uint64_t));
hipMemset(d_COUNT4, 0, sizeof(uint64_t));
count_5<<<std::ceil((float)nation_size/128.), 128>>>(d_COUNT4, nation_size);
uint64_t COUNT4;
hipMemcpy(&COUNT4, d_COUNT4, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_4;
hipMalloc(&d_BUF_IDX_4, sizeof(uint64_t));
hipMemset(d_BUF_IDX_4, 0, sizeof(uint64_t));
uint64_t* d_BUF_4;
hipMalloc(&d_BUF_4, sizeof(uint64_t) * COUNT4 * 1);
auto d_HT_4 = cuco::static_map{ (int)COUNT4*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5<<<std::ceil((float)nation_size/128.), 128>>>(d_BUF_4, d_BUF_IDX_4, d_HT_4.ref(cuco::insert), d_nation__n_nationkey, nation_size);
//Create aggregation hash table
auto d_HT_6 = cuco::static_map{ (int)466296*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_7<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_2, d_BUF_4, d_HT_2.ref(cuco::for_each), d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::insert), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
size_t COUNT6 = d_HT_6.size();
thrust::device_vector<int64_t> keys_6(COUNT6), vals_6(COUNT6);
d_HT_6.retrieve_all(keys_6.begin(), vals_6.begin());
d_HT_6.clear();
int64_t* raw_keys6 = thrust::raw_pointer_cast(keys_6.data());
insertKeys<<<std::ceil((float)COUNT6/128.), 128>>>(raw_keys6, d_HT_6.ref(cuco::insert), COUNT6);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT6);
DBI16Type* d_aggr__n_name_encoded;
hipMalloc(&d_aggr__n_name_encoded, sizeof(DBI16Type) * COUNT6);
hipMemset(d_aggr__n_name_encoded, 0, sizeof(DBI16Type) * COUNT6);
auto aggr__n_name_map = nation__n_name_map;
DBDecimalType* d_aggr__c_acctbal;
hipMalloc(&d_aggr__c_acctbal, sizeof(DBDecimalType) * COUNT6);
hipMemset(d_aggr__c_acctbal, 0, sizeof(DBDecimalType) * COUNT6);
DBI32Type* d_KEY_6customer__c_custkey;
hipMalloc(&d_KEY_6customer__c_custkey, sizeof(DBI32Type) * COUNT6);
hipMemset(d_KEY_6customer__c_custkey, 0, sizeof(DBI32Type) * COUNT6);
main_7<<<std::ceil((float)customer_size/128.), 128>>>(d_BUF_2, d_BUF_4, d_HT_2.ref(cuco::for_each), d_HT_4.ref(cuco::find), d_HT_6.ref(cuco::find), d_KEY_6customer__c_custkey, d_aggr0__tmp_attr0, d_aggr__c_acctbal, d_aggr__n_name_encoded, d_customer__c_acctbal, d_customer__c_custkey, d_customer__c_nationkey, customer_size, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_nation__n_name_encoded);
//Materialize count
uint64_t* d_COUNT8;
hipMalloc(&d_COUNT8, sizeof(uint64_t));
hipMemset(d_COUNT8, 0, sizeof(uint64_t));
count_9<<<std::ceil((float)COUNT6/128.), 128>>>(COUNT6, d_COUNT8);
uint64_t COUNT8;
hipMemcpy(&COUNT8, d_COUNT8, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX8;
hipMalloc(&d_MAT_IDX8, sizeof(uint64_t));
hipMemset(d_MAT_IDX8, 0, sizeof(uint64_t));
auto MAT8customer__c_custkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT8);
DBI32Type* d_MAT8customer__c_custkey;
hipMalloc(&d_MAT8customer__c_custkey, sizeof(DBI32Type) * COUNT8);
auto MAT8aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT8);
DBDecimalType* d_MAT8aggr0__tmp_attr0;
hipMalloc(&d_MAT8aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT8);
auto MAT8aggr__c_acctbal = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT8);
DBDecimalType* d_MAT8aggr__c_acctbal;
hipMalloc(&d_MAT8aggr__c_acctbal, sizeof(DBDecimalType) * COUNT8);
auto MAT8aggr__n_name_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT8);
DBI16Type* d_MAT8aggr__n_name_encoded;
hipMalloc(&d_MAT8aggr__n_name_encoded, sizeof(DBI16Type) * COUNT8);
main_9<<<std::ceil((float)COUNT6/128.), 128>>>(COUNT6, d_MAT8aggr0__tmp_attr0, d_MAT8aggr__c_acctbal, d_MAT8aggr__n_name_encoded, d_MAT8customer__c_custkey, d_MAT_IDX8, d_aggr0__tmp_attr0, d_aggr__c_acctbal, d_aggr__n_name_encoded, d_KEY_6customer__c_custkey);
hipMemcpy(MAT8customer__c_custkey, d_MAT8customer__c_custkey, sizeof(DBI32Type) * COUNT8, hipMemcpyDeviceToHost);
hipMemcpy(MAT8aggr0__tmp_attr0, d_MAT8aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT8, hipMemcpyDeviceToHost);
hipMemcpy(MAT8aggr__c_acctbal, d_MAT8aggr__c_acctbal, sizeof(DBDecimalType) * COUNT8, hipMemcpyDeviceToHost);
hipMemcpy(MAT8aggr__n_name_encoded, d_MAT8aggr__n_name_encoded, sizeof(DBI16Type) * COUNT8, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < COUNT8; i++) { std::cout << "" << MAT8customer__c_custkey[i];
std::cout << "|" << MAT8aggr0__tmp_attr0[i];
std::cout << "|" << MAT8aggr__c_acctbal[i];
std::cout << "|" << aggr__n_name_map[MAT8aggr__n_name_encoded[i]];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_COUNT0);
hipFree(d_BUF_2);
hipFree(d_BUF_IDX_2);
hipFree(d_COUNT2);
hipFree(d_BUF_4);
hipFree(d_BUF_IDX_4);
hipFree(d_COUNT4);
hipFree(d_KEY_6customer__c_custkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr__c_acctbal);
hipFree(d_aggr__n_name_encoded);
hipFree(d_COUNT8);
hipFree(d_MAT8aggr0__tmp_attr0);
hipFree(d_MAT8aggr__c_acctbal);
hipFree(d_MAT8aggr__n_name_encoded);
hipFree(d_MAT8customer__c_custkey);
hipFree(d_MAT_IDX8);
free(MAT8aggr0__tmp_attr0);
free(MAT8aggr__c_acctbal);
free(MAT8aggr__n_name_encoded);
free(MAT8customer__c_custkey);
}