#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
#include <chrono>
__global__ void count_1(uint64_t* COUNT0, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
//Materialize count
atomicAdd((int*)COUNT0, 1);
}
template<typename HASHTABLE_INSERT_PK>
__global__ void main_1(uint64_t* BUF_0, uint64_t* BUF_IDX_0, HASHTABLE_INSERT_PK HT_0, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_0 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_0 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_0 = atomicAdd((int*)BUF_IDX_0, 1);
HT_0.insert(cuco::pair{KEY_0, buf_idx_0});
BUF_0[buf_idx_0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_INSERT>
__global__ void count_3(uint64_t* BUF_0, HASHTABLE_PROBE_PK HT_0, HASHTABLE_INSERT HT_2, DBDateType* lineitem__l_commitdate, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_receiptdate, DBDateType* lineitem__l_shipdate, DBStringType* lineitem__l_shipmode, DBI16Type* lineitem__l_shipmode_encoded, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_receiptdate = lineitem__l_receiptdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_receiptdate, 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_receiptdate, 9131, Predicate::lt))) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
auto reg_lineitem__l_commitdate = lineitem__l_commitdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, reg_lineitem__l_commitdate, Predicate::lt))) return;
if (!(evaluatePredicate(reg_lineitem__l_commitdate, reg_lineitem__l_receiptdate, Predicate::lt))) return;
auto reg_lineitem__l_shipmode = lineitem__l_shipmode[tid];
if (!((evaluatePredicate(reg_lineitem__l_shipmode, "MAIL", Predicate::eq)) || (evaluatePredicate(reg_lineitem__l_shipmode, "SHIP", Predicate::eq)))) return;
uint64_t KEY_0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_0 |= reg_lineitem__l_orderkey;
//Probe Hash table
auto SLOT_0 = HT_0.find(KEY_0);
if (SLOT_0 == HT_0.end()) return;
if (!(true)) return;
uint64_t KEY_2 = 0;
auto reg_lineitem__l_shipmode_encoded = lineitem__l_shipmode_encoded[tid];

KEY_2 |= reg_lineitem__l_shipmode_encoded;
//Create aggregation hash table
HT_2.insert(cuco::pair{KEY_2, 1});
}
template<typename HASHTABLE_PROBE_PK, typename HASHTABLE_FIND>
__global__ void main_3(uint64_t* BUF_0, HASHTABLE_PROBE_PK HT_0, HASHTABLE_FIND HT_2, DBI16Type* KEY_2lineitem__l_shipmode_encoded, DBI32Type* aggr0__tmp_attr0, DBI32Type* aggr0__tmp_attr2, DBDateType* lineitem__l_commitdate, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_receiptdate, DBDateType* lineitem__l_shipdate, DBStringType* lineitem__l_shipmode, DBI16Type* lineitem__l_shipmode_encoded, size_t lineitem_size, DBStringType* orders__o_orderpriority) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_receiptdate = lineitem__l_receiptdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_receiptdate, 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_receiptdate, 9131, Predicate::lt))) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
auto reg_lineitem__l_commitdate = lineitem__l_commitdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, reg_lineitem__l_commitdate, Predicate::lt))) return;
if (!(evaluatePredicate(reg_lineitem__l_commitdate, reg_lineitem__l_receiptdate, Predicate::lt))) return;
auto reg_lineitem__l_shipmode = lineitem__l_shipmode[tid];
if (!((evaluatePredicate(reg_lineitem__l_shipmode, "MAIL", Predicate::eq)) || (evaluatePredicate(reg_lineitem__l_shipmode, "SHIP", Predicate::eq)))) return;
uint64_t KEY_0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_0 |= reg_lineitem__l_orderkey;
//Probe Hash table
auto SLOT_0 = HT_0.find(KEY_0);
if (SLOT_0 == HT_0.end()) return;
if (!(true)) return;
uint64_t KEY_2 = 0;
auto reg_lineitem__l_shipmode_encoded = lineitem__l_shipmode_encoded[tid];

KEY_2 |= reg_lineitem__l_shipmode_encoded;
//Aggregate in hashtable
auto buf_idx_2 = HT_2.find(KEY_2)->second;
auto reg_orders__o_orderpriority = orders__o_orderpriority[BUF_0[SLOT_0->second * 1 + 0]];
auto reg_map0__tmp_attr3 = (((evaluatePredicate(reg_orders__o_orderpriority, "1-URGENT", Predicate::neq)) && (evaluatePredicate(reg_orders__o_orderpriority, "2-HIGH", Predicate::neq))));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_2], reg_map0__tmp_attr3);
auto reg_map0__tmp_attr1 = (((evaluatePredicate(reg_orders__o_orderpriority, "1-URGENT", Predicate::eq)) || (evaluatePredicate(reg_orders__o_orderpriority, "2-HIGH", Predicate::eq))));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_2], reg_map0__tmp_attr1);
KEY_2lineitem__l_shipmode_encoded[buf_idx_2] = reg_lineitem__l_shipmode_encoded;
}
__global__ void count_5(size_t COUNT2, uint64_t* COUNT4) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT2) return;
//Materialize count
atomicAdd((int*)COUNT4, 1);
}
__global__ void main_5(size_t COUNT2, DBI32Type* MAT4aggr0__tmp_attr0, DBI32Type* MAT4aggr0__tmp_attr2, DBI16Type* MAT4lineitem__l_shipmode_encoded, uint64_t* MAT_IDX4, DBI32Type* aggr0__tmp_attr0, DBI32Type* aggr0__tmp_attr2, DBI16Type* lineitem__l_shipmode_encoded) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT2) return;
//Materialize buffers
auto mat_idx4 = atomicAdd((int*)MAT_IDX4, 1);
auto reg_lineitem__l_shipmode_encoded = lineitem__l_shipmode_encoded[tid];
MAT4lineitem__l_shipmode_encoded[mat_idx4] = reg_lineitem__l_shipmode_encoded;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT4aggr0__tmp_attr0[mat_idx4] = reg_aggr0__tmp_attr0;
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT4aggr0__tmp_attr2[mat_idx4] = reg_aggr0__tmp_attr2;
}
extern "C" void control (DBI32Type * d_nation__n_nationkey, DBStringType * d_nation__n_name, DBI32Type * d_nation__n_regionkey, DBStringType * d_nation__n_comment, size_t nation_size, DBI32Type * d_supplier__s_suppkey, DBI32Type * d_supplier__s_nationkey, DBStringType * d_supplier__s_name, DBStringType * d_supplier__s_address, DBStringType * d_supplier__s_phone, DBDecimalType * d_supplier__s_acctbal, DBStringType * d_supplier__s_comment, size_t supplier_size, DBI32Type * d_partsupp__ps_suppkey, DBI32Type * d_partsupp__ps_partkey, DBI32Type * d_partsupp__ps_availqty, DBDecimalType * d_partsupp__ps_supplycost, DBStringType * d_partsupp__ps_comment, size_t partsupp_size, DBI32Type * d_part__p_partkey, DBStringType * d_part__p_name, DBStringType * d_part__p_mfgr, DBStringType * d_part__p_brand, DBStringType * d_part__p_type, DBI32Type * d_part__p_size, DBStringType * d_part__p_container, DBDecimalType * d_part__p_retailprice, DBStringType * d_part__p_comment, size_t part_size, DBI32Type * d_lineitem__l_orderkey, DBI32Type * d_lineitem__l_partkey, DBI32Type * d_lineitem__l_suppkey, DBI64Type * d_lineitem__l_linenumber, DBDecimalType * d_lineitem__l_quantity, DBDecimalType * d_lineitem__l_extendedprice, DBDecimalType * d_lineitem__l_discount, DBDecimalType * d_lineitem__l_tax, DBCharType * d_lineitem__l_returnflag, DBCharType * d_lineitem__l_linestatus, DBI32Type * d_lineitem__l_shipdate, DBI32Type * d_lineitem__l_commitdate, DBI32Type * d_lineitem__l_receiptdate, DBStringType * d_lineitem__l_shipinstruct, DBStringType * d_lineitem__l_shipmode, DBStringType * d_lineitem__comments, size_t lineitem_size, DBI32Type * d_orders__o_orderkey, DBCharType * d_orders__o_orderstatus, DBI32Type * d_orders__o_custkey, DBDecimalType * d_orders__o_totalprice, DBI32Type * d_orders__o_orderdate, DBStringType * d_orders__o_orderpriority, DBStringType * d_orders__o_clerk, DBI32Type * d_orders__o_shippriority, DBStringType * d_orders__o_comment, size_t orders_size, DBI32Type * d_customer__c_custkey, DBStringType * d_customer__c_name, DBStringType * d_customer__c_address, DBI32Type * d_customer__c_nationkey, DBStringType * d_customer__c_phone, DBDecimalType * d_customer__c_acctbal, DBStringType * d_customer__c_mktsegment, DBStringType * d_customer__c_comment, size_t customer_size, DBI32Type * d_region__r_regionkey, DBStringType * d_region__r_name, DBStringType * d_region__r_comment, size_t region_size, DBI16Type* d_nation__n_name_encoded, std::unordered_map<DBI16Type, DBStringType> &nation__n_name_map, std::unordered_map<DBI16Type, DBStringType> &n1___n_name_map, std::unordered_map<DBI16Type, DBStringType> &n2___n_name_map, DBI16Type* d_orders__o_orderpriority_encoded, std::unordered_map<DBI16Type, std::string>& orders__o_orderpriority_map, DBI16Type* d_customer__c_name_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_name_map, DBI16Type* d_customer__c_comment_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_comment_map, DBI16Type* d_customer__c_phone_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_phone_map, DBI16Type* d_customer__c_address_encoded, std::unordered_map<DBI16Type, std::string>& customer__c_address_map, DBI16Type* d_supplier__s_name_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_name_map, DBI16Type* d_part__p_brand_encoded, std::unordered_map<DBI16Type, std::string>& part__p_brand_map, DBI16Type* d_part__p_type_encoded, std::unordered_map<DBI16Type, std::string>& part__p_type_map, DBI16Type* d_lineitem__l_shipmode_encoded, std::unordered_map<DBI16Type, std::string>& lineitem__l_shipmode_map, DBI16Type* d_supplier__s_address_encoded, std::unordered_map<DBI16Type, std::string>& supplier__s_address_map) {
size_t used_mem = usedGpuMem();
auto startTime = std::chrono::high_resolution_clock::now();
//Materialize count
uint64_t* d_COUNT0;
hipMalloc(&d_COUNT0, sizeof(uint64_t));
hipMemset(d_COUNT0, 0, sizeof(uint64_t));
count_1<<<std::ceil((float)orders_size/128.), 128>>>(d_COUNT0, orders_size);
uint64_t COUNT0;
hipMemcpy(&COUNT0, d_COUNT0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_0;
hipMalloc(&d_BUF_IDX_0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_0, 0, sizeof(uint64_t));
uint64_t* d_BUF_0;
hipMalloc(&d_BUF_0, sizeof(uint64_t) * COUNT0 * 1);
auto d_HT_0 = cuco::static_map{ (int)COUNT0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_1<<<std::ceil((float)orders_size/128.), 128>>>(d_BUF_0, d_BUF_IDX_0, d_HT_0.ref(cuco::insert), d_orders__o_orderkey, orders_size);
//Create aggregation hash table
auto d_HT_2 = cuco::static_map{ (int)292901*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_3<<<std::ceil((float)lineitem_size/128.), 128>>>(d_BUF_0, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::insert), d_lineitem__l_commitdate, d_lineitem__l_orderkey, d_lineitem__l_receiptdate, d_lineitem__l_shipdate, d_lineitem__l_shipmode, d_lineitem__l_shipmode_encoded, lineitem_size);
size_t COUNT2 = d_HT_2.size();
thrust::device_vector<int64_t> keys_2(COUNT2), vals_2(COUNT2);
d_HT_2.retrieve_all(keys_2.begin(), vals_2.begin());
d_HT_2.clear();
int64_t* raw_keys2 = thrust::raw_pointer_cast(keys_2.data());
insertKeys<<<std::ceil((float)COUNT2/128.), 128>>>(raw_keys2, d_HT_2.ref(cuco::insert), COUNT2);
//Aggregate in hashtable
DBI32Type* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBI32Type) * COUNT2);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBI32Type) * COUNT2);
DBI32Type* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBI32Type) * COUNT2);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBI32Type) * COUNT2);
DBI16Type* d_KEY_2lineitem__l_shipmode_encoded;
hipMalloc(&d_KEY_2lineitem__l_shipmode_encoded, sizeof(DBI16Type) * COUNT2);
hipMemset(d_KEY_2lineitem__l_shipmode_encoded, 0, sizeof(DBI16Type) * COUNT2);
main_3<<<std::ceil((float)lineitem_size/128.), 128>>>(d_BUF_0, d_HT_0.ref(cuco::find), d_HT_2.ref(cuco::find), d_KEY_2lineitem__l_shipmode_encoded, d_aggr0__tmp_attr0, d_aggr0__tmp_attr2, d_lineitem__l_commitdate, d_lineitem__l_orderkey, d_lineitem__l_receiptdate, d_lineitem__l_shipdate, d_lineitem__l_shipmode, d_lineitem__l_shipmode_encoded, lineitem_size, d_orders__o_orderpriority);
//Materialize count
uint64_t* d_COUNT4;
hipMalloc(&d_COUNT4, sizeof(uint64_t));
hipMemset(d_COUNT4, 0, sizeof(uint64_t));
count_5<<<std::ceil((float)COUNT2/128.), 128>>>(COUNT2, d_COUNT4);
uint64_t COUNT4;
hipMemcpy(&COUNT4, d_COUNT4, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX4;
hipMalloc(&d_MAT_IDX4, sizeof(uint64_t));
hipMemset(d_MAT_IDX4, 0, sizeof(uint64_t));
auto MAT4lineitem__l_shipmode_encoded = (DBI16Type*)malloc(sizeof(DBI16Type) * COUNT4);
DBI16Type* d_MAT4lineitem__l_shipmode_encoded;
hipMalloc(&d_MAT4lineitem__l_shipmode_encoded, sizeof(DBI16Type) * COUNT4);
auto MAT4aggr0__tmp_attr0 = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT4);
DBI32Type* d_MAT4aggr0__tmp_attr0;
hipMalloc(&d_MAT4aggr0__tmp_attr0, sizeof(DBI32Type) * COUNT4);
auto MAT4aggr0__tmp_attr2 = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT4);
DBI32Type* d_MAT4aggr0__tmp_attr2;
hipMalloc(&d_MAT4aggr0__tmp_attr2, sizeof(DBI32Type) * COUNT4);
main_5<<<std::ceil((float)COUNT2/128.), 128>>>(COUNT2, d_MAT4aggr0__tmp_attr0, d_MAT4aggr0__tmp_attr2, d_MAT4lineitem__l_shipmode_encoded, d_MAT_IDX4, d_aggr0__tmp_attr0, d_aggr0__tmp_attr2, d_KEY_2lineitem__l_shipmode_encoded);
hipMemcpy(MAT4lineitem__l_shipmode_encoded, d_MAT4lineitem__l_shipmode_encoded, sizeof(DBI16Type) * COUNT4, hipMemcpyDeviceToHost);
hipMemcpy(MAT4aggr0__tmp_attr0, d_MAT4aggr0__tmp_attr0, sizeof(DBI32Type) * COUNT4, hipMemcpyDeviceToHost);
hipMemcpy(MAT4aggr0__tmp_attr2, d_MAT4aggr0__tmp_attr2, sizeof(DBI32Type) * COUNT4, hipMemcpyDeviceToHost);
auto endTime = std::chrono::high_resolution_clock::now();
auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime);
std::clog << "Query execution time: " << duration.count() / 1000. << " milliseconds." << std::endl;

for (auto i=0ull; i < COUNT4; i++) { std::cout << "" << lineitem__l_shipmode_map[MAT4lineitem__l_shipmode_encoded[i]];
std::cout << "|" << MAT4aggr0__tmp_attr0[i];
std::cout << "|" << MAT4aggr0__tmp_attr2[i];
std::cout << std::endl; }
std::clog << "Used memory: " << used_mem / (1024 * 1024) << " MB" << std::endl; 
size_t aux_mem = usedGpuMem() - used_mem;
std::clog << "Auxiliary memory: " << aux_mem / (1024) << " KB" << std::endl;
hipFree(d_BUF_0);
hipFree(d_BUF_IDX_0);
hipFree(d_COUNT0);
hipFree(d_KEY_2lineitem__l_shipmode_encoded);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_COUNT4);
hipFree(d_MAT4aggr0__tmp_attr0);
hipFree(d_MAT4aggr0__tmp_attr2);
hipFree(d_MAT4lineitem__l_shipmode_encoded);
hipFree(d_MAT_IDX4);
free(MAT4aggr0__tmp_attr0);
free(MAT4aggr0__tmp_attr2);
free(MAT4lineitem__l_shipmode_encoded);
}