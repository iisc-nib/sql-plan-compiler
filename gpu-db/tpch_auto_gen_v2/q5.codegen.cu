#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_6476f24fdae0(uint64_t* COUNT6476f24ee790, DBStringType* region__r_name, size_t region_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= region_size) return;
auto reg_region__r_name = region__r_name[tid];
if (!(evaluatePredicate(reg_region__r_name, "ASIA", Predicate::eq))) return;
//Materialize count
atomicAdd((int*)COUNT6476f24ee790, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_6476f24fdae0(uint64_t* BUF_6476f24ee790, uint64_t* BUF_IDX_6476f24ee790, HASHTABLE_INSERT HT_6476f24ee790, DBStringType* region__r_name, DBI32Type* region__r_regionkey, size_t region_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= region_size) return;
auto reg_region__r_name = region__r_name[tid];
if (!(evaluatePredicate(reg_region__r_name, "ASIA", Predicate::eq))) return;
uint64_t KEY_6476f24ee790 = 0;
auto reg_region__r_regionkey = region__r_regionkey[tid];

KEY_6476f24ee790 |= reg_region__r_regionkey;
// Insert hash table kernel;
auto buf_idx_6476f24ee790 = atomicAdd((int*)BUF_IDX_6476f24ee790, 1);
HT_6476f24ee790.insert(cuco::pair{KEY_6476f24ee790, buf_idx_6476f24ee790});
BUF_6476f24ee790[buf_idx_6476f24ee790 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_6476f24fe9d0(uint64_t* BUF_6476f24ee790, uint64_t* COUNT6476f24f42c0, HASHTABLE_PROBE HT_6476f24ee790, DBI32Type* nation__n_regionkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_6476f24ee790 = 0;
auto reg_nation__n_regionkey = nation__n_regionkey[tid];

KEY_6476f24ee790 |= reg_nation__n_regionkey;
//Probe Hash table
HT_6476f24ee790.for_each(KEY_6476f24ee790, [&] __device__ (auto const SLOT_6476f24ee790) {

auto const [slot_first6476f24ee790, slot_second6476f24ee790] = SLOT_6476f24ee790;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT6476f24f42c0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_6476f24fe9d0(uint64_t* BUF_6476f24ee790, uint64_t* BUF_6476f24f42c0, uint64_t* BUF_IDX_6476f24f42c0, HASHTABLE_PROBE HT_6476f24ee790, HASHTABLE_INSERT HT_6476f24f42c0, DBI32Type* nation__n_nationkey, DBI32Type* nation__n_regionkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_6476f24ee790 = 0;
auto reg_nation__n_regionkey = nation__n_regionkey[tid];

KEY_6476f24ee790 |= reg_nation__n_regionkey;
//Probe Hash table
HT_6476f24ee790.for_each(KEY_6476f24ee790, [&] __device__ (auto const SLOT_6476f24ee790) {
auto const [slot_first6476f24ee790, slot_second6476f24ee790] = SLOT_6476f24ee790;
if (!(true)) return;
uint64_t KEY_6476f24f42c0 = 0;
auto reg_nation__n_nationkey = nation__n_nationkey[tid];

KEY_6476f24f42c0 |= reg_nation__n_nationkey;
// Insert hash table kernel;
auto buf_idx_6476f24f42c0 = atomicAdd((int*)BUF_IDX_6476f24f42c0, 1);
HT_6476f24f42c0.insert(cuco::pair{KEY_6476f24f42c0, buf_idx_6476f24f42c0});
BUF_6476f24f42c0[buf_idx_6476f24f42c0 * 2 + 0] = BUF_6476f24ee790[slot_second6476f24ee790 * 1 + 0];
BUF_6476f24f42c0[buf_idx_6476f24f42c0 * 2 + 1] = tid;
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_6476f24db130(uint64_t* BUF_6476f24f42c0, uint64_t* COUNT6476f24ee210, HASHTABLE_PROBE HT_6476f24f42c0, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_6476f24f42c0 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_6476f24f42c0 |= reg_customer__c_nationkey;
//Probe Hash table
HT_6476f24f42c0.for_each(KEY_6476f24f42c0, [&] __device__ (auto const SLOT_6476f24f42c0) {

auto const [slot_first6476f24f42c0, slot_second6476f24f42c0] = SLOT_6476f24f42c0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT6476f24ee210, 1);
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_6476f24db130(uint64_t* BUF_6476f24ee210, uint64_t* BUF_6476f24f42c0, uint64_t* BUF_IDX_6476f24ee210, HASHTABLE_INSERT HT_6476f24ee210, HASHTABLE_PROBE HT_6476f24f42c0, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_6476f24f42c0 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_6476f24f42c0 |= reg_customer__c_nationkey;
//Probe Hash table
HT_6476f24f42c0.for_each(KEY_6476f24f42c0, [&] __device__ (auto const SLOT_6476f24f42c0) {
auto const [slot_first6476f24f42c0, slot_second6476f24f42c0] = SLOT_6476f24f42c0;
if (!(true)) return;
uint64_t KEY_6476f24ee210 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_6476f24ee210 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_6476f24ee210 = atomicAdd((int*)BUF_IDX_6476f24ee210, 1);
HT_6476f24ee210.insert(cuco::pair{KEY_6476f24ee210, buf_idx_6476f24ee210});
BUF_6476f24ee210[buf_idx_6476f24ee210 * 3 + 0] = tid;
BUF_6476f24ee210[buf_idx_6476f24ee210 * 3 + 1] = BUF_6476f24f42c0[slot_second6476f24f42c0 * 2 + 0];
BUF_6476f24ee210[buf_idx_6476f24ee210 * 3 + 2] = BUF_6476f24f42c0[slot_second6476f24f42c0 * 2 + 1];
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_6476f24db6a0(uint64_t* BUF_6476f24ee210, uint64_t* COUNT6476f24f67b0, HASHTABLE_PROBE HT_6476f24ee210, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 9131, Predicate::lt))) return;
uint64_t KEY_6476f24ee210 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_6476f24ee210 |= reg_orders__o_custkey;
//Probe Hash table
HT_6476f24ee210.for_each(KEY_6476f24ee210, [&] __device__ (auto const SLOT_6476f24ee210) {

auto const [slot_first6476f24ee210, slot_second6476f24ee210] = SLOT_6476f24ee210;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT6476f24f67b0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_6476f24db6a0(uint64_t* BUF_6476f24ee210, uint64_t* BUF_6476f24f67b0, uint64_t* BUF_IDX_6476f24f67b0, HASHTABLE_PROBE HT_6476f24ee210, HASHTABLE_INSERT HT_6476f24f67b0, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 9131, Predicate::lt))) return;
uint64_t KEY_6476f24ee210 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_6476f24ee210 |= reg_orders__o_custkey;
//Probe Hash table
HT_6476f24ee210.for_each(KEY_6476f24ee210, [&] __device__ (auto const SLOT_6476f24ee210) {
auto const [slot_first6476f24ee210, slot_second6476f24ee210] = SLOT_6476f24ee210;
if (!(true)) return;
uint64_t KEY_6476f24f67b0 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_6476f24f67b0 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_6476f24f67b0 = atomicAdd((int*)BUF_IDX_6476f24f67b0, 1);
HT_6476f24f67b0.insert(cuco::pair{KEY_6476f24f67b0, buf_idx_6476f24f67b0});
BUF_6476f24f67b0[buf_idx_6476f24f67b0 * 4 + 0] = BUF_6476f24ee210[slot_second6476f24ee210 * 3 + 0];
BUF_6476f24f67b0[buf_idx_6476f24f67b0 * 4 + 1] = tid;
BUF_6476f24f67b0[buf_idx_6476f24f67b0 * 4 + 2] = BUF_6476f24ee210[slot_second6476f24ee210 * 3 + 1];
BUF_6476f24f67b0[buf_idx_6476f24f67b0 * 4 + 3] = BUF_6476f24ee210[slot_second6476f24ee210 * 3 + 2];
});
}
__global__ void count_6476f2507010(uint64_t* COUNT6476f24f68c0, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
//Materialize count
atomicAdd((int*)COUNT6476f24f68c0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_6476f2507010(uint64_t* BUF_6476f24f68c0, uint64_t* BUF_IDX_6476f24f68c0, HASHTABLE_INSERT HT_6476f24f68c0, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_6476f24f68c0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_6476f24f68c0 |= reg_supplier__s_suppkey;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];
KEY_6476f24f68c0 <<= 32;
KEY_6476f24f68c0 |= reg_supplier__s_nationkey;
// Insert hash table kernel;
auto buf_idx_6476f24f68c0 = atomicAdd((int*)BUF_IDX_6476f24f68c0, 1);
HT_6476f24f68c0.insert(cuco::pair{KEY_6476f24f68c0, buf_idx_6476f24f68c0});
BUF_6476f24f68c0[buf_idx_6476f24f68c0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_6476f24fb590(uint64_t* BUF_6476f24f67b0, uint64_t* BUF_6476f24f68c0, HASHTABLE_INSERT HT_6476f24aa320, HASHTABLE_PROBE HT_6476f24f67b0, HASHTABLE_PROBE HT_6476f24f68c0, DBI32Type* customer__c_nationkey, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_6476f24f67b0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_6476f24f67b0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_6476f24f67b0.for_each(KEY_6476f24f67b0, [&] __device__ (auto const SLOT_6476f24f67b0) {

auto const [slot_first6476f24f67b0, slot_second6476f24f67b0] = SLOT_6476f24f67b0;
if (!(true)) return;
uint64_t KEY_6476f24f68c0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_6476f24f68c0 |= reg_lineitem__l_suppkey;
auto reg_customer__c_nationkey = customer__c_nationkey[BUF_6476f24f67b0[slot_second6476f24f67b0 * 4 + 0]];
KEY_6476f24f68c0 <<= 32;
KEY_6476f24f68c0 |= reg_customer__c_nationkey;
//Probe Hash table
HT_6476f24f68c0.for_each(KEY_6476f24f68c0, [&] __device__ (auto const SLOT_6476f24f68c0) {

auto const [slot_first6476f24f68c0, slot_second6476f24f68c0] = SLOT_6476f24f68c0;
if (!(true)) return;
uint64_t KEY_6476f24aa320 = 0;
//Create aggregation hash table
HT_6476f24aa320.insert(cuco::pair{KEY_6476f24aa320, 1});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_6476f24fb590(uint64_t* BUF_6476f24f67b0, uint64_t* BUF_6476f24f68c0, HASHTABLE_FIND HT_6476f24aa320, HASHTABLE_PROBE HT_6476f24f67b0, HASHTABLE_PROBE HT_6476f24f68c0, DBDecimalType* aggr0__tmp_attr0, DBI32Type* customer__c_nationkey, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_6476f24f67b0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_6476f24f67b0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_6476f24f67b0.for_each(KEY_6476f24f67b0, [&] __device__ (auto const SLOT_6476f24f67b0) {
auto const [slot_first6476f24f67b0, slot_second6476f24f67b0] = SLOT_6476f24f67b0;
if (!(true)) return;
uint64_t KEY_6476f24f68c0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_6476f24f68c0 |= reg_lineitem__l_suppkey;
auto reg_customer__c_nationkey = customer__c_nationkey[BUF_6476f24f67b0[slot_second6476f24f67b0 * 4 + 0]];
KEY_6476f24f68c0 <<= 32;
KEY_6476f24f68c0 |= reg_customer__c_nationkey;
//Probe Hash table
HT_6476f24f68c0.for_each(KEY_6476f24f68c0, [&] __device__ (auto const SLOT_6476f24f68c0) {
auto const [slot_first6476f24f68c0, slot_second6476f24f68c0] = SLOT_6476f24f68c0;
if (!(true)) return;
uint64_t KEY_6476f24aa320 = 0;
//Aggregate in hashtable
auto buf_idx_6476f24aa320 = HT_6476f24aa320.find(KEY_6476f24aa320)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_6476f24aa320], reg_map0__tmp_attr1);
});
});
}
__global__ void count_6476f250f860(size_t COUNT6476f24aa320, uint64_t* COUNT6476f24bda90) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT6476f24aa320) return;
//Materialize count
atomicAdd((int*)COUNT6476f24bda90, 1);
}
__global__ void main_6476f250f860(size_t COUNT6476f24aa320, DBDecimalType* MAT6476f24bda90aggr0__tmp_attr0, uint64_t* MAT_IDX6476f24bda90, DBDecimalType* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT6476f24aa320) return;
//Materialize buffers
auto mat_idx6476f24bda90 = atomicAdd((int*)MAT_IDX6476f24bda90, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT6476f24bda90aggr0__tmp_attr0[mat_idx6476f24bda90] = reg_aggr0__tmp_attr0;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT6476f24ee790;
hipMalloc(&d_COUNT6476f24ee790, sizeof(uint64_t));
hipMemset(d_COUNT6476f24ee790, 0, sizeof(uint64_t));
count_6476f24fdae0<<<std::ceil((float)region_size/32.), 32>>>(d_COUNT6476f24ee790, d_region__r_name, region_size);
uint64_t COUNT6476f24ee790;
hipMemcpy(&COUNT6476f24ee790, d_COUNT6476f24ee790, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_6476f24ee790;
hipMalloc(&d_BUF_IDX_6476f24ee790, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6476f24ee790, 0, sizeof(uint64_t));
uint64_t* d_BUF_6476f24ee790;
hipMalloc(&d_BUF_6476f24ee790, sizeof(uint64_t) * COUNT6476f24ee790 * 1);
auto d_HT_6476f24ee790 = cuco::experimental::static_multimap{ (int)COUNT6476f24ee790*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_6476f24fdae0<<<std::ceil((float)region_size/32.), 32>>>(d_BUF_6476f24ee790, d_BUF_IDX_6476f24ee790, d_HT_6476f24ee790.ref(cuco::insert), d_region__r_name, d_region__r_regionkey, region_size);
//Materialize count
uint64_t* d_COUNT6476f24f42c0;
hipMalloc(&d_COUNT6476f24f42c0, sizeof(uint64_t));
hipMemset(d_COUNT6476f24f42c0, 0, sizeof(uint64_t));
count_6476f24fe9d0<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_6476f24ee790, d_COUNT6476f24f42c0, d_HT_6476f24ee790.ref(cuco::for_each), d_nation__n_regionkey, nation_size);
uint64_t COUNT6476f24f42c0;
hipMemcpy(&COUNT6476f24f42c0, d_COUNT6476f24f42c0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_6476f24f42c0;
hipMalloc(&d_BUF_IDX_6476f24f42c0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6476f24f42c0, 0, sizeof(uint64_t));
uint64_t* d_BUF_6476f24f42c0;
hipMalloc(&d_BUF_6476f24f42c0, sizeof(uint64_t) * COUNT6476f24f42c0 * 2);
auto d_HT_6476f24f42c0 = cuco::experimental::static_multimap{ (int)COUNT6476f24f42c0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_6476f24fe9d0<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_6476f24ee790, d_BUF_6476f24f42c0, d_BUF_IDX_6476f24f42c0, d_HT_6476f24ee790.ref(cuco::for_each), d_HT_6476f24f42c0.ref(cuco::insert), d_nation__n_nationkey, d_nation__n_regionkey, nation_size);
//Materialize count
uint64_t* d_COUNT6476f24ee210;
hipMalloc(&d_COUNT6476f24ee210, sizeof(uint64_t));
hipMemset(d_COUNT6476f24ee210, 0, sizeof(uint64_t));
count_6476f24db130<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_6476f24f42c0, d_COUNT6476f24ee210, d_HT_6476f24f42c0.ref(cuco::for_each), d_customer__c_nationkey, customer_size);
uint64_t COUNT6476f24ee210;
hipMemcpy(&COUNT6476f24ee210, d_COUNT6476f24ee210, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_6476f24ee210;
hipMalloc(&d_BUF_IDX_6476f24ee210, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6476f24ee210, 0, sizeof(uint64_t));
uint64_t* d_BUF_6476f24ee210;
hipMalloc(&d_BUF_6476f24ee210, sizeof(uint64_t) * COUNT6476f24ee210 * 3);
auto d_HT_6476f24ee210 = cuco::experimental::static_multimap{ (int)COUNT6476f24ee210*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_6476f24db130<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_6476f24ee210, d_BUF_6476f24f42c0, d_BUF_IDX_6476f24ee210, d_HT_6476f24ee210.ref(cuco::insert), d_HT_6476f24f42c0.ref(cuco::for_each), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
//Materialize count
uint64_t* d_COUNT6476f24f67b0;
hipMalloc(&d_COUNT6476f24f67b0, sizeof(uint64_t));
hipMemset(d_COUNT6476f24f67b0, 0, sizeof(uint64_t));
count_6476f24db6a0<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_6476f24ee210, d_COUNT6476f24f67b0, d_HT_6476f24ee210.ref(cuco::for_each), d_orders__o_custkey, d_orders__o_orderdate, orders_size);
uint64_t COUNT6476f24f67b0;
hipMemcpy(&COUNT6476f24f67b0, d_COUNT6476f24f67b0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_6476f24f67b0;
hipMalloc(&d_BUF_IDX_6476f24f67b0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6476f24f67b0, 0, sizeof(uint64_t));
uint64_t* d_BUF_6476f24f67b0;
hipMalloc(&d_BUF_6476f24f67b0, sizeof(uint64_t) * COUNT6476f24f67b0 * 4);
auto d_HT_6476f24f67b0 = cuco::experimental::static_multimap{ (int)COUNT6476f24f67b0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_6476f24db6a0<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_6476f24ee210, d_BUF_6476f24f67b0, d_BUF_IDX_6476f24f67b0, d_HT_6476f24ee210.ref(cuco::for_each), d_HT_6476f24f67b0.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
//Materialize count
uint64_t* d_COUNT6476f24f68c0;
hipMalloc(&d_COUNT6476f24f68c0, sizeof(uint64_t));
hipMemset(d_COUNT6476f24f68c0, 0, sizeof(uint64_t));
count_6476f2507010<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT6476f24f68c0, supplier_size);
uint64_t COUNT6476f24f68c0;
hipMemcpy(&COUNT6476f24f68c0, d_COUNT6476f24f68c0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_6476f24f68c0;
hipMalloc(&d_BUF_IDX_6476f24f68c0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6476f24f68c0, 0, sizeof(uint64_t));
uint64_t* d_BUF_6476f24f68c0;
hipMalloc(&d_BUF_6476f24f68c0, sizeof(uint64_t) * COUNT6476f24f68c0 * 1);
auto d_HT_6476f24f68c0 = cuco::experimental::static_multimap{ (int)COUNT6476f24f68c0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_6476f2507010<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_6476f24f68c0, d_BUF_IDX_6476f24f68c0, d_HT_6476f24f68c0.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
//Create aggregation hash table
auto d_HT_6476f24aa320 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_6476f24fb590<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_6476f24f67b0, d_BUF_6476f24f68c0, d_HT_6476f24aa320.ref(cuco::insert), d_HT_6476f24f67b0.ref(cuco::for_each), d_HT_6476f24f68c0.ref(cuco::for_each), d_customer__c_nationkey, d_lineitem__l_orderkey, d_lineitem__l_suppkey, lineitem_size);
size_t COUNT6476f24aa320 = d_HT_6476f24aa320.size();
thrust::device_vector<int64_t> keys_6476f24aa320(COUNT6476f24aa320), vals_6476f24aa320(COUNT6476f24aa320);
d_HT_6476f24aa320.retrieve_all(keys_6476f24aa320.begin(), vals_6476f24aa320.begin());
d_HT_6476f24aa320.clear();
int64_t* raw_keys6476f24aa320 = thrust::raw_pointer_cast(keys_6476f24aa320.data());
insertKeys<<<std::ceil((float)COUNT6476f24aa320/32.), 32>>>(raw_keys6476f24aa320, d_HT_6476f24aa320.ref(cuco::insert), COUNT6476f24aa320);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6476f24aa320);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT6476f24aa320);
main_6476f24fb590<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_6476f24f67b0, d_BUF_6476f24f68c0, d_HT_6476f24aa320.ref(cuco::find), d_HT_6476f24f67b0.ref(cuco::for_each), d_HT_6476f24f68c0.ref(cuco::for_each), d_aggr0__tmp_attr0, d_customer__c_nationkey, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_suppkey, lineitem_size);
//Materialize count
uint64_t* d_COUNT6476f24bda90;
hipMalloc(&d_COUNT6476f24bda90, sizeof(uint64_t));
hipMemset(d_COUNT6476f24bda90, 0, sizeof(uint64_t));
count_6476f250f860<<<std::ceil((float)COUNT6476f24aa320/32.), 32>>>(COUNT6476f24aa320, d_COUNT6476f24bda90);
uint64_t COUNT6476f24bda90;
hipMemcpy(&COUNT6476f24bda90, d_COUNT6476f24bda90, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX6476f24bda90;
hipMalloc(&d_MAT_IDX6476f24bda90, sizeof(uint64_t));
hipMemset(d_MAT_IDX6476f24bda90, 0, sizeof(uint64_t));
auto MAT6476f24bda90aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT6476f24bda90);
DBDecimalType* d_MAT6476f24bda90aggr0__tmp_attr0;
hipMalloc(&d_MAT6476f24bda90aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6476f24bda90);
main_6476f250f860<<<std::ceil((float)COUNT6476f24aa320/32.), 32>>>(COUNT6476f24aa320, d_MAT6476f24bda90aggr0__tmp_attr0, d_MAT_IDX6476f24bda90, d_aggr0__tmp_attr0);
hipMemcpy(MAT6476f24bda90aggr0__tmp_attr0, d_MAT6476f24bda90aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6476f24bda90, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT6476f24bda90; i++) { std::cout << MAT6476f24bda90aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_6476f24ee790);
hipFree(d_BUF_IDX_6476f24ee790);
hipFree(d_COUNT6476f24ee790);
hipFree(d_BUF_6476f24f42c0);
hipFree(d_BUF_IDX_6476f24f42c0);
hipFree(d_COUNT6476f24f42c0);
hipFree(d_BUF_6476f24ee210);
hipFree(d_BUF_IDX_6476f24ee210);
hipFree(d_COUNT6476f24ee210);
hipFree(d_BUF_6476f24f67b0);
hipFree(d_BUF_IDX_6476f24f67b0);
hipFree(d_COUNT6476f24f67b0);
hipFree(d_BUF_6476f24f68c0);
hipFree(d_BUF_IDX_6476f24f68c0);
hipFree(d_COUNT6476f24f68c0);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT6476f24bda90);
hipFree(d_MAT6476f24bda90aggr0__tmp_attr0);
hipFree(d_MAT_IDX6476f24bda90);
free(MAT6476f24bda90aggr0__tmp_attr0);
}