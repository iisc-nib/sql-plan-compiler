#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_6064ce1e8a40(uint64_t* COUNT6064ce1d81b0, DBStringType* region__r_name, size_t region_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= region_size) return;
auto reg_region__r_name = region__r_name[tid];
if (!(evaluatePredicate(reg_region__r_name, "ASIA", Predicate::eq))) return;
//Materialize count
atomicAdd((int*)COUNT6064ce1d81b0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_6064ce1e8a40(uint64_t* BUF_6064ce1d81b0, uint64_t* BUF_IDX_6064ce1d81b0, HASHTABLE_INSERT HT_6064ce1d81b0, DBStringType* region__r_name, DBI32Type* region__r_regionkey, size_t region_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= region_size) return;
auto reg_region__r_name = region__r_name[tid];
if (!(evaluatePredicate(reg_region__r_name, "ASIA", Predicate::eq))) return;
uint64_t KEY_6064ce1d81b0 = 0;
auto reg_region__r_regionkey = region__r_regionkey[tid];

KEY_6064ce1d81b0 |= reg_region__r_regionkey;
// Insert hash table kernel;
auto buf_idx_6064ce1d81b0 = atomicAdd((int*)BUF_IDX_6064ce1d81b0, 1);
HT_6064ce1d81b0.insert(cuco::pair{KEY_6064ce1d81b0, buf_idx_6064ce1d81b0});
BUF_6064ce1d81b0[buf_idx_6064ce1d81b0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_6064ce1e98d0(uint64_t* BUF_6064ce1d81b0, uint64_t* COUNT6064ce1d84a0, HASHTABLE_PROBE HT_6064ce1d81b0, DBI32Type* nation__n_regionkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_6064ce1d81b0 = 0;
auto reg_nation__n_regionkey = nation__n_regionkey[tid];

KEY_6064ce1d81b0 |= reg_nation__n_regionkey;
//Probe Hash table
HT_6064ce1d81b0.for_each(KEY_6064ce1d81b0, [&] __device__ (auto const SLOT_6064ce1d81b0) {

auto const [slot_first6064ce1d81b0, slot_second6064ce1d81b0] = SLOT_6064ce1d81b0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT6064ce1d84a0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_6064ce1e98d0(uint64_t* BUF_6064ce1d81b0, uint64_t* BUF_6064ce1d84a0, uint64_t* BUF_IDX_6064ce1d84a0, HASHTABLE_PROBE HT_6064ce1d81b0, HASHTABLE_INSERT HT_6064ce1d84a0, DBI32Type* nation__n_nationkey, DBI32Type* nation__n_regionkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_6064ce1d81b0 = 0;
auto reg_nation__n_regionkey = nation__n_regionkey[tid];

KEY_6064ce1d81b0 |= reg_nation__n_regionkey;
//Probe Hash table
HT_6064ce1d81b0.for_each(KEY_6064ce1d81b0, [&] __device__ (auto const SLOT_6064ce1d81b0) {
auto const [slot_first6064ce1d81b0, slot_second6064ce1d81b0] = SLOT_6064ce1d81b0;
if (!(true)) return;
uint64_t KEY_6064ce1d84a0 = 0;
auto reg_nation__n_nationkey = nation__n_nationkey[tid];

KEY_6064ce1d84a0 |= reg_nation__n_nationkey;
// Insert hash table kernel;
auto buf_idx_6064ce1d84a0 = atomicAdd((int*)BUF_IDX_6064ce1d84a0, 1);
HT_6064ce1d84a0.insert(cuco::pair{KEY_6064ce1d84a0, buf_idx_6064ce1d84a0});
BUF_6064ce1d84a0[buf_idx_6064ce1d84a0 * 2 + 0] = BUF_6064ce1d81b0[slot_second6064ce1d81b0 * 1 + 0];
BUF_6064ce1d84a0[buf_idx_6064ce1d84a0 * 2 + 1] = tid;
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_6064ce0dedd0(uint64_t* BUF_6064ce1d84a0, uint64_t* COUNT6064ce1e1480, HASHTABLE_PROBE HT_6064ce1d84a0, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_6064ce1d84a0 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_6064ce1d84a0 |= reg_customer__c_nationkey;
//Probe Hash table
HT_6064ce1d84a0.for_each(KEY_6064ce1d84a0, [&] __device__ (auto const SLOT_6064ce1d84a0) {

auto const [slot_first6064ce1d84a0, slot_second6064ce1d84a0] = SLOT_6064ce1d84a0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT6064ce1e1480, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_6064ce0dedd0(uint64_t* BUF_6064ce1d84a0, uint64_t* BUF_6064ce1e1480, uint64_t* BUF_IDX_6064ce1e1480, HASHTABLE_PROBE HT_6064ce1d84a0, HASHTABLE_INSERT HT_6064ce1e1480, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_6064ce1d84a0 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_6064ce1d84a0 |= reg_customer__c_nationkey;
//Probe Hash table
HT_6064ce1d84a0.for_each(KEY_6064ce1d84a0, [&] __device__ (auto const SLOT_6064ce1d84a0) {
auto const [slot_first6064ce1d84a0, slot_second6064ce1d84a0] = SLOT_6064ce1d84a0;
if (!(true)) return;
uint64_t KEY_6064ce1e1480 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_6064ce1e1480 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_6064ce1e1480 = atomicAdd((int*)BUF_IDX_6064ce1e1480, 1);
HT_6064ce1e1480.insert(cuco::pair{KEY_6064ce1e1480, buf_idx_6064ce1e1480});
BUF_6064ce1e1480[buf_idx_6064ce1e1480 * 3 + 0] = tid;
BUF_6064ce1e1480[buf_idx_6064ce1e1480 * 3 + 1] = BUF_6064ce1d84a0[slot_second6064ce1d84a0 * 2 + 0];
BUF_6064ce1e1480[buf_idx_6064ce1e1480 * 3 + 2] = BUF_6064ce1d84a0[slot_second6064ce1d84a0 * 2 + 1];
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_6064ce162140(uint64_t* BUF_6064ce1e1480, uint64_t* COUNT6064ce1e15f0, HASHTABLE_PROBE HT_6064ce1e1480, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 9131, Predicate::lt))) return;
uint64_t KEY_6064ce1e1480 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_6064ce1e1480 |= reg_orders__o_custkey;
//Probe Hash table
HT_6064ce1e1480.for_each(KEY_6064ce1e1480, [&] __device__ (auto const SLOT_6064ce1e1480) {

auto const [slot_first6064ce1e1480, slot_second6064ce1e1480] = SLOT_6064ce1e1480;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT6064ce1e15f0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_6064ce162140(uint64_t* BUF_6064ce1e1480, uint64_t* BUF_6064ce1e15f0, uint64_t* BUF_IDX_6064ce1e15f0, HASHTABLE_PROBE HT_6064ce1e1480, HASHTABLE_INSERT HT_6064ce1e15f0, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 9131, Predicate::lt))) return;
uint64_t KEY_6064ce1e1480 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_6064ce1e1480 |= reg_orders__o_custkey;
//Probe Hash table
HT_6064ce1e1480.for_each(KEY_6064ce1e1480, [&] __device__ (auto const SLOT_6064ce1e1480) {
auto const [slot_first6064ce1e1480, slot_second6064ce1e1480] = SLOT_6064ce1e1480;
if (!(true)) return;
uint64_t KEY_6064ce1e15f0 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_6064ce1e15f0 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_6064ce1e15f0 = atomicAdd((int*)BUF_IDX_6064ce1e15f0, 1);
HT_6064ce1e15f0.insert(cuco::pair{KEY_6064ce1e15f0, buf_idx_6064ce1e15f0});
BUF_6064ce1e15f0[buf_idx_6064ce1e15f0 * 4 + 0] = BUF_6064ce1e1480[slot_second6064ce1e1480 * 3 + 0];
BUF_6064ce1e15f0[buf_idx_6064ce1e15f0 * 4 + 1] = tid;
BUF_6064ce1e15f0[buf_idx_6064ce1e15f0 * 4 + 2] = BUF_6064ce1e1480[slot_second6064ce1e1480 * 3 + 1];
BUF_6064ce1e15f0[buf_idx_6064ce1e15f0 * 4 + 3] = BUF_6064ce1e1480[slot_second6064ce1e1480 * 3 + 2];
});
}
__global__ void count_6064ce1f1ad0(uint64_t* COUNT6064ce1e1700, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
//Materialize count
atomicAdd((int*)COUNT6064ce1e1700, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_6064ce1f1ad0(uint64_t* BUF_6064ce1e1700, uint64_t* BUF_IDX_6064ce1e1700, HASHTABLE_INSERT HT_6064ce1e1700, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_6064ce1e1700 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_6064ce1e1700 |= reg_supplier__s_suppkey;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];
KEY_6064ce1e1700 <<= 32;
KEY_6064ce1e1700 |= reg_supplier__s_nationkey;
// Insert hash table kernel;
auto buf_idx_6064ce1e1700 = atomicAdd((int*)BUF_IDX_6064ce1e1700, 1);
HT_6064ce1e1700.insert(cuco::pair{KEY_6064ce1e1700, buf_idx_6064ce1e1700});
BUF_6064ce1e1700[buf_idx_6064ce1e1700 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_6064ce1e6780(uint64_t* BUF_6064ce1e15f0, uint64_t* BUF_6064ce1e1700, HASHTABLE_INSERT HT_6064ce195100, HASHTABLE_PROBE HT_6064ce1e15f0, HASHTABLE_PROBE HT_6064ce1e1700, DBI32Type* customer__c_nationkey, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_6064ce1e15f0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_6064ce1e15f0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_6064ce1e15f0.for_each(KEY_6064ce1e15f0, [&] __device__ (auto const SLOT_6064ce1e15f0) {

auto const [slot_first6064ce1e15f0, slot_second6064ce1e15f0] = SLOT_6064ce1e15f0;
if (!(true)) return;
uint64_t KEY_6064ce1e1700 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_6064ce1e1700 |= reg_lineitem__l_suppkey;
auto reg_customer__c_nationkey = customer__c_nationkey[BUF_6064ce1e15f0[slot_second6064ce1e15f0 * 4 + 0]];
KEY_6064ce1e1700 <<= 32;
KEY_6064ce1e1700 |= reg_customer__c_nationkey;
//Probe Hash table
HT_6064ce1e1700.for_each(KEY_6064ce1e1700, [&] __device__ (auto const SLOT_6064ce1e1700) {

auto const [slot_first6064ce1e1700, slot_second6064ce1e1700] = SLOT_6064ce1e1700;
if (!(true)) return;
uint64_t KEY_6064ce195100 = 0;
//Create aggregation hash table
HT_6064ce195100.insert(cuco::pair{KEY_6064ce195100, 1});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_6064ce1e6780(uint64_t* BUF_6064ce1e15f0, uint64_t* BUF_6064ce1e1700, HASHTABLE_FIND HT_6064ce195100, HASHTABLE_PROBE HT_6064ce1e15f0, HASHTABLE_PROBE HT_6064ce1e1700, DBDecimalType* aggr0__tmp_attr0, DBI32Type* customer__c_nationkey, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_6064ce1e15f0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_6064ce1e15f0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_6064ce1e15f0.for_each(KEY_6064ce1e15f0, [&] __device__ (auto const SLOT_6064ce1e15f0) {
auto const [slot_first6064ce1e15f0, slot_second6064ce1e15f0] = SLOT_6064ce1e15f0;
if (!(true)) return;
uint64_t KEY_6064ce1e1700 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_6064ce1e1700 |= reg_lineitem__l_suppkey;
auto reg_customer__c_nationkey = customer__c_nationkey[BUF_6064ce1e15f0[slot_second6064ce1e15f0 * 4 + 0]];
KEY_6064ce1e1700 <<= 32;
KEY_6064ce1e1700 |= reg_customer__c_nationkey;
//Probe Hash table
HT_6064ce1e1700.for_each(KEY_6064ce1e1700, [&] __device__ (auto const SLOT_6064ce1e1700) {
auto const [slot_first6064ce1e1700, slot_second6064ce1e1700] = SLOT_6064ce1e1700;
if (!(true)) return;
uint64_t KEY_6064ce195100 = 0;
//Aggregate in hashtable
auto buf_idx_6064ce195100 = HT_6064ce195100.find(KEY_6064ce195100)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_6064ce195100], reg_map0__tmp_attr1);
});
});
}
__global__ void count_6064ce1faed0(size_t COUNT6064ce195100, uint64_t* COUNT6064ce1a7fa0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT6064ce195100) return;
//Materialize count
atomicAdd((int*)COUNT6064ce1a7fa0, 1);
}
__global__ void main_6064ce1faed0(size_t COUNT6064ce195100, DBDecimalType* MAT6064ce1a7fa0aggr0__tmp_attr0, uint64_t* MAT_IDX6064ce1a7fa0, DBDecimalType* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT6064ce195100) return;
//Materialize buffers
auto mat_idx6064ce1a7fa0 = atomicAdd((int*)MAT_IDX6064ce1a7fa0, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT6064ce1a7fa0aggr0__tmp_attr0[mat_idx6064ce1a7fa0] = reg_aggr0__tmp_attr0;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT6064ce1d81b0;
hipMalloc(&d_COUNT6064ce1d81b0, sizeof(uint64_t));
hipMemset(d_COUNT6064ce1d81b0, 0, sizeof(uint64_t));
count_6064ce1e8a40<<<std::ceil((float)region_size/32.), 32>>>(d_COUNT6064ce1d81b0, d_region__r_name, region_size);
uint64_t COUNT6064ce1d81b0;
hipMemcpy(&COUNT6064ce1d81b0, d_COUNT6064ce1d81b0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT6064ce1d81b0);
// Insert hash table control;
uint64_t* d_BUF_IDX_6064ce1d81b0;
hipMalloc(&d_BUF_IDX_6064ce1d81b0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6064ce1d81b0, 0, sizeof(uint64_t));
uint64_t* d_BUF_6064ce1d81b0;
hipMalloc(&d_BUF_6064ce1d81b0, sizeof(uint64_t) * COUNT6064ce1d81b0 * 1);
auto d_HT_6064ce1d81b0 = cuco::experimental::static_multimap{ (int)COUNT6064ce1d81b0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_6064ce1e8a40<<<std::ceil((float)region_size/32.), 32>>>(d_BUF_6064ce1d81b0, d_BUF_IDX_6064ce1d81b0, d_HT_6064ce1d81b0.ref(cuco::insert), d_region__r_name, d_region__r_regionkey, region_size);
hipFree(d_BUF_IDX_6064ce1d81b0);
//Materialize count
uint64_t* d_COUNT6064ce1d84a0;
hipMalloc(&d_COUNT6064ce1d84a0, sizeof(uint64_t));
hipMemset(d_COUNT6064ce1d84a0, 0, sizeof(uint64_t));
count_6064ce1e98d0<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_6064ce1d81b0, d_COUNT6064ce1d84a0, d_HT_6064ce1d81b0.ref(cuco::for_each), d_nation__n_regionkey, nation_size);
uint64_t COUNT6064ce1d84a0;
hipMemcpy(&COUNT6064ce1d84a0, d_COUNT6064ce1d84a0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT6064ce1d84a0);
// Insert hash table control;
uint64_t* d_BUF_IDX_6064ce1d84a0;
hipMalloc(&d_BUF_IDX_6064ce1d84a0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6064ce1d84a0, 0, sizeof(uint64_t));
uint64_t* d_BUF_6064ce1d84a0;
hipMalloc(&d_BUF_6064ce1d84a0, sizeof(uint64_t) * COUNT6064ce1d84a0 * 2);
auto d_HT_6064ce1d84a0 = cuco::experimental::static_multimap{ (int)COUNT6064ce1d84a0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_6064ce1e98d0<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_6064ce1d81b0, d_BUF_6064ce1d84a0, d_BUF_IDX_6064ce1d84a0, d_HT_6064ce1d81b0.ref(cuco::for_each), d_HT_6064ce1d84a0.ref(cuco::insert), d_nation__n_nationkey, d_nation__n_regionkey, nation_size);
hipFree(d_BUF_IDX_6064ce1d84a0);
//Materialize count
uint64_t* d_COUNT6064ce1e1480;
hipMalloc(&d_COUNT6064ce1e1480, sizeof(uint64_t));
hipMemset(d_COUNT6064ce1e1480, 0, sizeof(uint64_t));
count_6064ce0dedd0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_6064ce1d84a0, d_COUNT6064ce1e1480, d_HT_6064ce1d84a0.ref(cuco::for_each), d_customer__c_nationkey, customer_size);
uint64_t COUNT6064ce1e1480;
hipMemcpy(&COUNT6064ce1e1480, d_COUNT6064ce1e1480, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT6064ce1e1480);
// Insert hash table control;
uint64_t* d_BUF_IDX_6064ce1e1480;
hipMalloc(&d_BUF_IDX_6064ce1e1480, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6064ce1e1480, 0, sizeof(uint64_t));
uint64_t* d_BUF_6064ce1e1480;
hipMalloc(&d_BUF_6064ce1e1480, sizeof(uint64_t) * COUNT6064ce1e1480 * 3);
auto d_HT_6064ce1e1480 = cuco::experimental::static_multimap{ (int)COUNT6064ce1e1480*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_6064ce0dedd0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_6064ce1d84a0, d_BUF_6064ce1e1480, d_BUF_IDX_6064ce1e1480, d_HT_6064ce1d84a0.ref(cuco::for_each), d_HT_6064ce1e1480.ref(cuco::insert), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
hipFree(d_BUF_IDX_6064ce1e1480);
//Materialize count
uint64_t* d_COUNT6064ce1e15f0;
hipMalloc(&d_COUNT6064ce1e15f0, sizeof(uint64_t));
hipMemset(d_COUNT6064ce1e15f0, 0, sizeof(uint64_t));
count_6064ce162140<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_6064ce1e1480, d_COUNT6064ce1e15f0, d_HT_6064ce1e1480.ref(cuco::for_each), d_orders__o_custkey, d_orders__o_orderdate, orders_size);
uint64_t COUNT6064ce1e15f0;
hipMemcpy(&COUNT6064ce1e15f0, d_COUNT6064ce1e15f0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT6064ce1e15f0);
// Insert hash table control;
uint64_t* d_BUF_IDX_6064ce1e15f0;
hipMalloc(&d_BUF_IDX_6064ce1e15f0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6064ce1e15f0, 0, sizeof(uint64_t));
uint64_t* d_BUF_6064ce1e15f0;
hipMalloc(&d_BUF_6064ce1e15f0, sizeof(uint64_t) * COUNT6064ce1e15f0 * 4);
auto d_HT_6064ce1e15f0 = cuco::experimental::static_multimap{ (int)COUNT6064ce1e15f0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_6064ce162140<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_6064ce1e1480, d_BUF_6064ce1e15f0, d_BUF_IDX_6064ce1e15f0, d_HT_6064ce1e1480.ref(cuco::for_each), d_HT_6064ce1e15f0.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
hipFree(d_BUF_IDX_6064ce1e15f0);
//Materialize count
uint64_t* d_COUNT6064ce1e1700;
hipMalloc(&d_COUNT6064ce1e1700, sizeof(uint64_t));
hipMemset(d_COUNT6064ce1e1700, 0, sizeof(uint64_t));
count_6064ce1f1ad0<<<std::ceil((float)supplier_size/32.), 32>>>(d_COUNT6064ce1e1700, supplier_size);
uint64_t COUNT6064ce1e1700;
hipMemcpy(&COUNT6064ce1e1700, d_COUNT6064ce1e1700, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT6064ce1e1700);
// Insert hash table control;
uint64_t* d_BUF_IDX_6064ce1e1700;
hipMalloc(&d_BUF_IDX_6064ce1e1700, sizeof(uint64_t));
hipMemset(d_BUF_IDX_6064ce1e1700, 0, sizeof(uint64_t));
uint64_t* d_BUF_6064ce1e1700;
hipMalloc(&d_BUF_6064ce1e1700, sizeof(uint64_t) * COUNT6064ce1e1700 * 1);
auto d_HT_6064ce1e1700 = cuco::experimental::static_multimap{ (int)COUNT6064ce1e1700*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_6064ce1f1ad0<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_6064ce1e1700, d_BUF_IDX_6064ce1e1700, d_HT_6064ce1e1700.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
hipFree(d_BUF_IDX_6064ce1e1700);
//Create aggregation hash table
auto d_HT_6064ce195100 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_6064ce1e6780<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_6064ce1e15f0, d_BUF_6064ce1e1700, d_HT_6064ce195100.ref(cuco::insert), d_HT_6064ce1e15f0.ref(cuco::for_each), d_HT_6064ce1e1700.ref(cuco::for_each), d_customer__c_nationkey, d_lineitem__l_orderkey, d_lineitem__l_suppkey, lineitem_size);
size_t COUNT6064ce195100 = d_HT_6064ce195100.size();
thrust::device_vector<int64_t> keys_6064ce195100(COUNT6064ce195100), vals_6064ce195100(COUNT6064ce195100);
d_HT_6064ce195100.retrieve_all(keys_6064ce195100.begin(), vals_6064ce195100.begin());
d_HT_6064ce195100.clear();
int64_t* raw_keys6064ce195100 = thrust::raw_pointer_cast(keys_6064ce195100.data());
insertKeys<<<std::ceil((float)COUNT6064ce195100/32.), 32>>>(raw_keys6064ce195100, d_HT_6064ce195100.ref(cuco::insert), COUNT6064ce195100);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6064ce195100);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT6064ce195100);
main_6064ce1e6780<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_6064ce1e15f0, d_BUF_6064ce1e1700, d_HT_6064ce195100.ref(cuco::find), d_HT_6064ce1e15f0.ref(cuco::for_each), d_HT_6064ce1e1700.ref(cuco::for_each), d_aggr0__tmp_attr0, d_customer__c_nationkey, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_suppkey, lineitem_size);
//Materialize count
uint64_t* d_COUNT6064ce1a7fa0;
hipMalloc(&d_COUNT6064ce1a7fa0, sizeof(uint64_t));
hipMemset(d_COUNT6064ce1a7fa0, 0, sizeof(uint64_t));
count_6064ce1faed0<<<std::ceil((float)COUNT6064ce195100/32.), 32>>>(COUNT6064ce195100, d_COUNT6064ce1a7fa0);
uint64_t COUNT6064ce1a7fa0;
hipMemcpy(&COUNT6064ce1a7fa0, d_COUNT6064ce1a7fa0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT6064ce1a7fa0);
//Materialize buffers
uint64_t* d_MAT_IDX6064ce1a7fa0;
hipMalloc(&d_MAT_IDX6064ce1a7fa0, sizeof(uint64_t));
hipMemset(d_MAT_IDX6064ce1a7fa0, 0, sizeof(uint64_t));
auto MAT6064ce1a7fa0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT6064ce1a7fa0);
DBDecimalType* d_MAT6064ce1a7fa0aggr0__tmp_attr0;
hipMalloc(&d_MAT6064ce1a7fa0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6064ce1a7fa0);
main_6064ce1faed0<<<std::ceil((float)COUNT6064ce195100/32.), 32>>>(COUNT6064ce195100, d_MAT6064ce1a7fa0aggr0__tmp_attr0, d_MAT_IDX6064ce1a7fa0, d_aggr0__tmp_attr0);
hipFree(d_MAT_IDX6064ce1a7fa0);
hipMemcpy(MAT6064ce1a7fa0aggr0__tmp_attr0, d_MAT6064ce1a7fa0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6064ce1a7fa0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT6064ce1a7fa0; i++) { std::cout << MAT6064ce1a7fa0aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
}