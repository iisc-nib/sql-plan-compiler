#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_591de844c170(uint64_t* COUNT591de8429310, DBStringType* n1___n_name, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n1___n_name = n1___n_name[tid];
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
//Materialize count
atomicAdd((int*)COUNT591de8429310, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_591de844c170(uint64_t* BUF_591de8429310, uint64_t* BUF_IDX_591de8429310, HASHTABLE_INSERT HT_591de8429310, DBStringType* n1___n_name, DBI32Type* n1___n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n1___n_name = n1___n_name[tid];
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
uint64_t KEY_591de8429310 = 0;
auto reg_n1___n_nationkey = n1___n_nationkey[tid];

KEY_591de8429310 |= reg_n1___n_nationkey;
// Insert hash table kernel;
auto buf_idx_591de8429310 = atomicAdd((int*)BUF_IDX_591de8429310, 1);
HT_591de8429310.insert(cuco::pair{KEY_591de8429310, buf_idx_591de8429310});
BUF_591de8429310[buf_idx_591de8429310 * 1 + 0] = tid;
}
__global__ void count_591de844f3c0(uint64_t* COUNT591de8440c60, DBStringType* n2___n_name, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n2___n_name = n2___n_name[tid];
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
//Materialize count
atomicAdd((int*)COUNT591de8440c60, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_591de844f3c0(uint64_t* BUF_591de8440c60, uint64_t* BUF_IDX_591de8440c60, HASHTABLE_INSERT HT_591de8440c60, DBStringType* n2___n_name, DBI32Type* n2___n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n2___n_name = n2___n_name[tid];
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
uint64_t KEY_591de8440c60 = 0;
auto reg_n2___n_nationkey = n2___n_nationkey[tid];

KEY_591de8440c60 |= reg_n2___n_nationkey;
// Insert hash table kernel;
auto buf_idx_591de8440c60 = atomicAdd((int*)BUF_IDX_591de8440c60, 1);
HT_591de8440c60.insert(cuco::pair{KEY_591de8440c60, buf_idx_591de8440c60});
BUF_591de8440c60[buf_idx_591de8440c60 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_591de8450320(uint64_t* BUF_591de8440c60, uint64_t* COUNT591de843d4e0, HASHTABLE_PROBE HT_591de8440c60, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_591de8440c60 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_591de8440c60 |= reg_customer__c_nationkey;
//Probe Hash table
HT_591de8440c60.for_each(KEY_591de8440c60, [&] __device__ (auto const SLOT_591de8440c60) {

auto const [slot_first591de8440c60, slot_second591de8440c60] = SLOT_591de8440c60;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT591de843d4e0, 1);
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_591de8450320(uint64_t* BUF_591de843d4e0, uint64_t* BUF_591de8440c60, uint64_t* BUF_IDX_591de843d4e0, HASHTABLE_INSERT HT_591de843d4e0, HASHTABLE_PROBE HT_591de8440c60, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_591de8440c60 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_591de8440c60 |= reg_customer__c_nationkey;
//Probe Hash table
HT_591de8440c60.for_each(KEY_591de8440c60, [&] __device__ (auto const SLOT_591de8440c60) {
auto const [slot_first591de8440c60, slot_second591de8440c60] = SLOT_591de8440c60;
if (!(true)) return;
uint64_t KEY_591de843d4e0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_591de843d4e0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_591de843d4e0 = atomicAdd((int*)BUF_IDX_591de843d4e0, 1);
HT_591de843d4e0.insert(cuco::pair{KEY_591de843d4e0, buf_idx_591de843d4e0});
BUF_591de843d4e0[buf_idx_591de843d4e0 * 2 + 0] = BUF_591de8440c60[slot_second591de8440c60 * 1 + 0];
BUF_591de843d4e0[buf_idx_591de843d4e0 * 2 + 1] = tid;
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_591de844ad20(uint64_t* BUF_591de843d4e0, uint64_t* COUNT591de8441dd0, HASHTABLE_PROBE HT_591de843d4e0, DBI32Type* orders__o_custkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_591de843d4e0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_591de843d4e0 |= reg_orders__o_custkey;
//Probe Hash table
HT_591de843d4e0.for_each(KEY_591de843d4e0, [&] __device__ (auto const SLOT_591de843d4e0) {

auto const [slot_first591de843d4e0, slot_second591de843d4e0] = SLOT_591de843d4e0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT591de8441dd0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_591de844ad20(uint64_t* BUF_591de843d4e0, uint64_t* BUF_591de8441dd0, uint64_t* BUF_IDX_591de8441dd0, HASHTABLE_PROBE HT_591de843d4e0, HASHTABLE_INSERT HT_591de8441dd0, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_591de843d4e0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_591de843d4e0 |= reg_orders__o_custkey;
//Probe Hash table
HT_591de843d4e0.for_each(KEY_591de843d4e0, [&] __device__ (auto const SLOT_591de843d4e0) {
auto const [slot_first591de843d4e0, slot_second591de843d4e0] = SLOT_591de843d4e0;
if (!(true)) return;
uint64_t KEY_591de8441dd0 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_591de8441dd0 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_591de8441dd0 = atomicAdd((int*)BUF_IDX_591de8441dd0, 1);
HT_591de8441dd0.insert(cuco::pair{KEY_591de8441dd0, buf_idx_591de8441dd0});
BUF_591de8441dd0[buf_idx_591de8441dd0 * 3 + 0] = tid;
BUF_591de8441dd0[buf_idx_591de8441dd0 * 3 + 1] = BUF_591de843d4e0[slot_second591de843d4e0 * 2 + 0];
BUF_591de8441dd0[buf_idx_591de8441dd0 * 3 + 2] = BUF_591de843d4e0[slot_second591de843d4e0 * 2 + 1];
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_591de83412c0(uint64_t* BUF_591de8429310, uint64_t* COUNT591de8441ee0, HASHTABLE_PROBE HT_591de8429310, DBI32Type* supplier__s_nationkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_591de8429310 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_591de8429310 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_591de8429310.for_each(KEY_591de8429310, [&] __device__ (auto const SLOT_591de8429310) {

auto const [slot_first591de8429310, slot_second591de8429310] = SLOT_591de8429310;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT591de8441ee0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_591de83412c0(uint64_t* BUF_591de8429310, uint64_t* BUF_591de8441ee0, uint64_t* BUF_IDX_591de8441ee0, HASHTABLE_PROBE HT_591de8429310, HASHTABLE_INSERT HT_591de8441ee0, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_591de8429310 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_591de8429310 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_591de8429310.for_each(KEY_591de8429310, [&] __device__ (auto const SLOT_591de8429310) {
auto const [slot_first591de8429310, slot_second591de8429310] = SLOT_591de8429310;
if (!(true)) return;
uint64_t KEY_591de8441ee0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_591de8441ee0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_591de8441ee0 = atomicAdd((int*)BUF_IDX_591de8441ee0, 1);
HT_591de8441ee0.insert(cuco::pair{KEY_591de8441ee0, buf_idx_591de8441ee0});
BUF_591de8441ee0[buf_idx_591de8441ee0 * 2 + 0] = tid;
BUF_591de8441ee0[buf_idx_591de8441ee0 * 2 + 1] = BUF_591de8429310[slot_second591de8429310 * 1 + 0];
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_591de83c4880(uint64_t* BUF_591de8441dd0, uint64_t* BUF_591de8441ee0, HASHTABLE_INSERT HT_591de83e4300, HASHTABLE_PROBE HT_591de8441dd0, HASHTABLE_PROBE HT_591de8441ee0, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n1___n_name, DBStringType* n2___n_name) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9861, Predicate::lte))) return;
uint64_t KEY_591de8441dd0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_591de8441dd0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_591de8441dd0.for_each(KEY_591de8441dd0, [&] __device__ (auto const SLOT_591de8441dd0) {

auto const [slot_first591de8441dd0, slot_second591de8441dd0] = SLOT_591de8441dd0;
if (!(true)) return;
uint64_t KEY_591de8441ee0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_591de8441ee0 |= reg_lineitem__l_suppkey;
//Probe Hash table
HT_591de8441ee0.for_each(KEY_591de8441ee0, [&] __device__ (auto const SLOT_591de8441ee0) {

auto const [slot_first591de8441ee0, slot_second591de8441ee0] = SLOT_591de8441ee0;
auto reg_n1___n_name = n1___n_name[BUF_591de8441ee0[slot_second591de8441ee0 * 2 + 1]];
auto reg_n2___n_name = n2___n_name[BUF_591de8441dd0[slot_second591de8441dd0 * 3 + 1]];
if (!((((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq))) || ((evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) && (true))) return;
uint64_t KEY_591de83e4300 = 0;
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_lineitem__l_shipdate);

KEY_591de83e4300 |= reg_map0__tmp_attr0;
//Create aggregation hash table
HT_591de83e4300.insert(cuco::pair{KEY_591de83e4300, 1});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_591de83c4880(uint64_t* BUF_591de8441dd0, uint64_t* BUF_591de8441ee0, HASHTABLE_FIND HT_591de83e4300, HASHTABLE_PROBE HT_591de8441dd0, HASHTABLE_PROBE HT_591de8441ee0, DBI64Type* KEY_591de83e4300map0__tmp_attr0, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n1___n_name, DBStringType* n2___n_name) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9861, Predicate::lte))) return;
uint64_t KEY_591de8441dd0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_591de8441dd0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_591de8441dd0.for_each(KEY_591de8441dd0, [&] __device__ (auto const SLOT_591de8441dd0) {
auto const [slot_first591de8441dd0, slot_second591de8441dd0] = SLOT_591de8441dd0;
if (!(true)) return;
uint64_t KEY_591de8441ee0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_591de8441ee0 |= reg_lineitem__l_suppkey;
//Probe Hash table
HT_591de8441ee0.for_each(KEY_591de8441ee0, [&] __device__ (auto const SLOT_591de8441ee0) {
auto const [slot_first591de8441ee0, slot_second591de8441ee0] = SLOT_591de8441ee0;
auto reg_n1___n_name = n1___n_name[BUF_591de8441ee0[slot_second591de8441ee0 * 2 + 1]];
auto reg_n2___n_name = n2___n_name[BUF_591de8441dd0[slot_second591de8441dd0 * 3 + 1]];
if (!((((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq))) || ((evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) && (true))) return;
uint64_t KEY_591de83e4300 = 0;
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_lineitem__l_shipdate);

KEY_591de83e4300 |= reg_map0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_591de83e4300 = HT_591de83e4300.find(KEY_591de83e4300)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_591de83e4300], reg_map0__tmp_attr1);
KEY_591de83e4300map0__tmp_attr0[buf_idx_591de83e4300] = reg_map0__tmp_attr0;
});
});
}
__global__ void count_591de8460c40(size_t COUNT591de83e4300, uint64_t* COUNT591de840aac0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT591de83e4300) return;
//Materialize count
atomicAdd((int*)COUNT591de840aac0, 1);
}
__global__ void main_591de8460c40(size_t COUNT591de83e4300, DBDecimalType* MAT591de840aac0aggr0__tmp_attr2, DBI64Type* MAT591de840aac0map0__tmp_attr0, uint64_t* MAT_IDX591de840aac0, DBDecimalType* aggr0__tmp_attr2, DBI64Type* map0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT591de83e4300) return;
//Materialize buffers
auto mat_idx591de840aac0 = atomicAdd((int*)MAT_IDX591de840aac0, 1);
auto reg_map0__tmp_attr0 = map0__tmp_attr0[tid];
MAT591de840aac0map0__tmp_attr0[mat_idx591de840aac0] = reg_map0__tmp_attr0;
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT591de840aac0aggr0__tmp_attr2[mat_idx591de840aac0] = reg_aggr0__tmp_attr2;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT591de8429310;
hipMalloc(&d_COUNT591de8429310, sizeof(uint64_t));
hipMemset(d_COUNT591de8429310, 0, sizeof(uint64_t));
count_591de844c170<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT591de8429310, d_nation__n_name, nation_size);
uint64_t COUNT591de8429310;
hipMemcpy(&COUNT591de8429310, d_COUNT591de8429310, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT591de8429310);
// Insert hash table control;
uint64_t* d_BUF_IDX_591de8429310;
hipMalloc(&d_BUF_IDX_591de8429310, sizeof(uint64_t));
hipMemset(d_BUF_IDX_591de8429310, 0, sizeof(uint64_t));
uint64_t* d_BUF_591de8429310;
hipMalloc(&d_BUF_591de8429310, sizeof(uint64_t) * COUNT591de8429310 * 1);
auto d_HT_591de8429310 = cuco::experimental::static_multimap{ (int)COUNT591de8429310*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_591de844c170<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_591de8429310, d_BUF_IDX_591de8429310, d_HT_591de8429310.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
hipFree(d_BUF_IDX_591de8429310);
//Materialize count
uint64_t* d_COUNT591de8440c60;
hipMalloc(&d_COUNT591de8440c60, sizeof(uint64_t));
hipMemset(d_COUNT591de8440c60, 0, sizeof(uint64_t));
count_591de844f3c0<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT591de8440c60, d_nation__n_name, nation_size);
uint64_t COUNT591de8440c60;
hipMemcpy(&COUNT591de8440c60, d_COUNT591de8440c60, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT591de8440c60);
// Insert hash table control;
uint64_t* d_BUF_IDX_591de8440c60;
hipMalloc(&d_BUF_IDX_591de8440c60, sizeof(uint64_t));
hipMemset(d_BUF_IDX_591de8440c60, 0, sizeof(uint64_t));
uint64_t* d_BUF_591de8440c60;
hipMalloc(&d_BUF_591de8440c60, sizeof(uint64_t) * COUNT591de8440c60 * 1);
auto d_HT_591de8440c60 = cuco::experimental::static_multimap{ (int)COUNT591de8440c60*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_591de844f3c0<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_591de8440c60, d_BUF_IDX_591de8440c60, d_HT_591de8440c60.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
hipFree(d_BUF_IDX_591de8440c60);
//Materialize count
uint64_t* d_COUNT591de843d4e0;
hipMalloc(&d_COUNT591de843d4e0, sizeof(uint64_t));
hipMemset(d_COUNT591de843d4e0, 0, sizeof(uint64_t));
count_591de8450320<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_591de8440c60, d_COUNT591de843d4e0, d_HT_591de8440c60.ref(cuco::for_each), d_customer__c_nationkey, customer_size);
uint64_t COUNT591de843d4e0;
hipMemcpy(&COUNT591de843d4e0, d_COUNT591de843d4e0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT591de843d4e0);
// Insert hash table control;
uint64_t* d_BUF_IDX_591de843d4e0;
hipMalloc(&d_BUF_IDX_591de843d4e0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_591de843d4e0, 0, sizeof(uint64_t));
uint64_t* d_BUF_591de843d4e0;
hipMalloc(&d_BUF_591de843d4e0, sizeof(uint64_t) * COUNT591de843d4e0 * 2);
auto d_HT_591de843d4e0 = cuco::experimental::static_multimap{ (int)COUNT591de843d4e0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_591de8450320<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_591de843d4e0, d_BUF_591de8440c60, d_BUF_IDX_591de843d4e0, d_HT_591de843d4e0.ref(cuco::insert), d_HT_591de8440c60.ref(cuco::for_each), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
hipFree(d_BUF_IDX_591de843d4e0);
//Materialize count
uint64_t* d_COUNT591de8441dd0;
hipMalloc(&d_COUNT591de8441dd0, sizeof(uint64_t));
hipMemset(d_COUNT591de8441dd0, 0, sizeof(uint64_t));
count_591de844ad20<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_591de843d4e0, d_COUNT591de8441dd0, d_HT_591de843d4e0.ref(cuco::for_each), d_orders__o_custkey, orders_size);
uint64_t COUNT591de8441dd0;
hipMemcpy(&COUNT591de8441dd0, d_COUNT591de8441dd0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT591de8441dd0);
// Insert hash table control;
uint64_t* d_BUF_IDX_591de8441dd0;
hipMalloc(&d_BUF_IDX_591de8441dd0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_591de8441dd0, 0, sizeof(uint64_t));
uint64_t* d_BUF_591de8441dd0;
hipMalloc(&d_BUF_591de8441dd0, sizeof(uint64_t) * COUNT591de8441dd0 * 3);
auto d_HT_591de8441dd0 = cuco::experimental::static_multimap{ (int)COUNT591de8441dd0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_591de844ad20<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_591de843d4e0, d_BUF_591de8441dd0, d_BUF_IDX_591de8441dd0, d_HT_591de843d4e0.ref(cuco::for_each), d_HT_591de8441dd0.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderkey, orders_size);
hipFree(d_BUF_IDX_591de8441dd0);
//Materialize count
uint64_t* d_COUNT591de8441ee0;
hipMalloc(&d_COUNT591de8441ee0, sizeof(uint64_t));
hipMemset(d_COUNT591de8441ee0, 0, sizeof(uint64_t));
count_591de83412c0<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_591de8429310, d_COUNT591de8441ee0, d_HT_591de8429310.ref(cuco::for_each), d_supplier__s_nationkey, supplier_size);
uint64_t COUNT591de8441ee0;
hipMemcpy(&COUNT591de8441ee0, d_COUNT591de8441ee0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT591de8441ee0);
// Insert hash table control;
uint64_t* d_BUF_IDX_591de8441ee0;
hipMalloc(&d_BUF_IDX_591de8441ee0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_591de8441ee0, 0, sizeof(uint64_t));
uint64_t* d_BUF_591de8441ee0;
hipMalloc(&d_BUF_591de8441ee0, sizeof(uint64_t) * COUNT591de8441ee0 * 2);
auto d_HT_591de8441ee0 = cuco::experimental::static_multimap{ (int)COUNT591de8441ee0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_591de83412c0<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_591de8429310, d_BUF_591de8441ee0, d_BUF_IDX_591de8441ee0, d_HT_591de8429310.ref(cuco::for_each), d_HT_591de8441ee0.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
hipFree(d_BUF_IDX_591de8441ee0);
//Create aggregation hash table
auto d_HT_591de83e4300 = cuco::static_map{ (int)13634*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_591de83c4880<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_591de8441dd0, d_BUF_591de8441ee0, d_HT_591de83e4300.ref(cuco::insert), d_HT_591de8441dd0.ref(cuco::for_each), d_HT_591de8441ee0.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_nation__n_name);
size_t COUNT591de83e4300 = d_HT_591de83e4300.size();
thrust::device_vector<int64_t> keys_591de83e4300(COUNT591de83e4300), vals_591de83e4300(COUNT591de83e4300);
d_HT_591de83e4300.retrieve_all(keys_591de83e4300.begin(), vals_591de83e4300.begin());
thrust::host_vector<int64_t> h_keys_591de83e4300(COUNT591de83e4300);
thrust::copy(keys_591de83e4300.begin(), keys_591de83e4300.end(), h_keys_591de83e4300.begin());
thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_591de83e4300(COUNT591de83e4300);
for (int i=0; i < COUNT591de83e4300; i++)
{actual_dict_591de83e4300[i] = cuco::make_pair(h_keys_591de83e4300[i], i);}
d_HT_591de83e4300.clear();
d_HT_591de83e4300.insert(actual_dict_591de83e4300.begin(), actual_dict_591de83e4300.end());
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT591de83e4300);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT591de83e4300);
DBI64Type* d_KEY_591de83e4300map0__tmp_attr0;
hipMalloc(&d_KEY_591de83e4300map0__tmp_attr0, sizeof(DBI64Type) * COUNT591de83e4300);
hipMemset(d_KEY_591de83e4300map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT591de83e4300);
main_591de83c4880<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_591de8441dd0, d_BUF_591de8441ee0, d_HT_591de83e4300.ref(cuco::find), d_HT_591de8441dd0.ref(cuco::for_each), d_HT_591de8441ee0.ref(cuco::for_each), d_KEY_591de83e4300map0__tmp_attr0, d_aggr0__tmp_attr2, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_nation__n_name);
//Materialize count
uint64_t* d_COUNT591de840aac0;
hipMalloc(&d_COUNT591de840aac0, sizeof(uint64_t));
hipMemset(d_COUNT591de840aac0, 0, sizeof(uint64_t));
count_591de8460c40<<<std::ceil((float)COUNT591de83e4300/32.), 32>>>(COUNT591de83e4300, d_COUNT591de840aac0);
uint64_t COUNT591de840aac0;
hipMemcpy(&COUNT591de840aac0, d_COUNT591de840aac0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT591de840aac0);
//Materialize buffers
uint64_t* d_MAT_IDX591de840aac0;
hipMalloc(&d_MAT_IDX591de840aac0, sizeof(uint64_t));
hipMemset(d_MAT_IDX591de840aac0, 0, sizeof(uint64_t));
auto MAT591de840aac0map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT591de840aac0);
DBI64Type* d_MAT591de840aac0map0__tmp_attr0;
hipMalloc(&d_MAT591de840aac0map0__tmp_attr0, sizeof(DBI64Type) * COUNT591de840aac0);
auto MAT591de840aac0aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT591de840aac0);
DBDecimalType* d_MAT591de840aac0aggr0__tmp_attr2;
hipMalloc(&d_MAT591de840aac0aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT591de840aac0);
main_591de8460c40<<<std::ceil((float)COUNT591de83e4300/32.), 32>>>(COUNT591de83e4300, d_MAT591de840aac0aggr0__tmp_attr2, d_MAT591de840aac0map0__tmp_attr0, d_MAT_IDX591de840aac0, d_aggr0__tmp_attr2, d_KEY_591de83e4300map0__tmp_attr0);
hipFree(d_MAT_IDX591de840aac0);
hipMemcpy(MAT591de840aac0map0__tmp_attr0, d_MAT591de840aac0map0__tmp_attr0, sizeof(DBI64Type) * COUNT591de840aac0, hipMemcpyDeviceToHost);
hipMemcpy(MAT591de840aac0aggr0__tmp_attr2, d_MAT591de840aac0aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT591de840aac0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT591de840aac0; i++) { std::cout << MAT591de840aac0map0__tmp_attr0[i] << "\t";
std::cout << MAT591de840aac0aggr0__tmp_attr2[i] << "\t";
std::cout << std::endl; }
}