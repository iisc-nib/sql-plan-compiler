#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5b4bb43a76a0(uint64_t* COUNT5b4bb439aa50, DBStringType* n1___n_name, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n1___n_name = n1___n_name[tid];
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
//Materialize count
atomicAdd((int*)COUNT5b4bb439aa50, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5b4bb43a76a0(uint64_t* BUF_5b4bb439aa50, uint64_t* BUF_IDX_5b4bb439aa50, HASHTABLE_INSERT HT_5b4bb439aa50, DBStringType* n1___n_name, DBI32Type* n1___n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n1___n_name = n1___n_name[tid];
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
uint64_t KEY_5b4bb439aa50 = 0;
auto reg_n1___n_nationkey = n1___n_nationkey[tid];

KEY_5b4bb439aa50 |= reg_n1___n_nationkey;
// Insert hash table kernel;
auto buf_idx_5b4bb439aa50 = atomicAdd((int*)BUF_IDX_5b4bb439aa50, 1);
HT_5b4bb439aa50.insert(cuco::pair{KEY_5b4bb439aa50, buf_idx_5b4bb439aa50});
BUF_5b4bb439aa50[buf_idx_5b4bb439aa50 * 1 + 0] = tid;
}
__global__ void count_5b4bb43aa820(uint64_t* COUNT5b4bb4399bf0, DBStringType* n2___n_name, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n2___n_name = n2___n_name[tid];
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
//Materialize count
atomicAdd((int*)COUNT5b4bb4399bf0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5b4bb43aa820(uint64_t* BUF_5b4bb4399bf0, uint64_t* BUF_IDX_5b4bb4399bf0, HASHTABLE_INSERT HT_5b4bb4399bf0, DBStringType* n2___n_name, DBI32Type* n2___n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n2___n_name = n2___n_name[tid];
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
uint64_t KEY_5b4bb4399bf0 = 0;
auto reg_n2___n_nationkey = n2___n_nationkey[tid];

KEY_5b4bb4399bf0 |= reg_n2___n_nationkey;
// Insert hash table kernel;
auto buf_idx_5b4bb4399bf0 = atomicAdd((int*)BUF_IDX_5b4bb4399bf0, 1);
HT_5b4bb4399bf0.insert(cuco::pair{KEY_5b4bb4399bf0, buf_idx_5b4bb4399bf0});
BUF_5b4bb4399bf0[buf_idx_5b4bb4399bf0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5b4bb43ab780(uint64_t* BUF_5b4bb4399bf0, uint64_t* COUNT5b4bb4398620, HASHTABLE_PROBE HT_5b4bb4399bf0, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_5b4bb4399bf0 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_5b4bb4399bf0 |= reg_customer__c_nationkey;
//Probe Hash table
HT_5b4bb4399bf0.for_each(KEY_5b4bb4399bf0, [&] __device__ (auto const SLOT_5b4bb4399bf0) {

auto const [slot_first5b4bb4399bf0, slot_second5b4bb4399bf0] = SLOT_5b4bb4399bf0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5b4bb4398620, 1);
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_5b4bb43ab780(uint64_t* BUF_5b4bb4398620, uint64_t* BUF_5b4bb4399bf0, uint64_t* BUF_IDX_5b4bb4398620, HASHTABLE_INSERT HT_5b4bb4398620, HASHTABLE_PROBE HT_5b4bb4399bf0, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_5b4bb4399bf0 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_5b4bb4399bf0 |= reg_customer__c_nationkey;
//Probe Hash table
HT_5b4bb4399bf0.for_each(KEY_5b4bb4399bf0, [&] __device__ (auto const SLOT_5b4bb4399bf0) {
auto const [slot_first5b4bb4399bf0, slot_second5b4bb4399bf0] = SLOT_5b4bb4399bf0;
if (!(true)) return;
uint64_t KEY_5b4bb4398620 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5b4bb4398620 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_5b4bb4398620 = atomicAdd((int*)BUF_IDX_5b4bb4398620, 1);
HT_5b4bb4398620.insert(cuco::pair{KEY_5b4bb4398620, buf_idx_5b4bb4398620});
BUF_5b4bb4398620[buf_idx_5b4bb4398620 * 2 + 0] = BUF_5b4bb4399bf0[slot_second5b4bb4399bf0 * 1 + 0];
BUF_5b4bb4398620[buf_idx_5b4bb4398620 * 2 + 1] = tid;
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_5b4bb43a6160(uint64_t* BUF_5b4bb4398620, uint64_t* COUNT5b4bb4398d20, HASHTABLE_PROBE HT_5b4bb4398620, DBI32Type* orders__o_custkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_5b4bb4398620 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_5b4bb4398620 |= reg_orders__o_custkey;
//Probe Hash table
HT_5b4bb4398620.for_each(KEY_5b4bb4398620, [&] __device__ (auto const SLOT_5b4bb4398620) {

auto const [slot_first5b4bb4398620, slot_second5b4bb4398620] = SLOT_5b4bb4398620;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5b4bb4398d20, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5b4bb43a6160(uint64_t* BUF_5b4bb4398620, uint64_t* BUF_5b4bb4398d20, uint64_t* BUF_IDX_5b4bb4398d20, HASHTABLE_PROBE HT_5b4bb4398620, HASHTABLE_INSERT HT_5b4bb4398d20, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_5b4bb4398620 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_5b4bb4398620 |= reg_orders__o_custkey;
//Probe Hash table
HT_5b4bb4398620.for_each(KEY_5b4bb4398620, [&] __device__ (auto const SLOT_5b4bb4398620) {
auto const [slot_first5b4bb4398620, slot_second5b4bb4398620] = SLOT_5b4bb4398620;
if (!(true)) return;
uint64_t KEY_5b4bb4398d20 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_5b4bb4398d20 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_5b4bb4398d20 = atomicAdd((int*)BUF_IDX_5b4bb4398d20, 1);
HT_5b4bb4398d20.insert(cuco::pair{KEY_5b4bb4398d20, buf_idx_5b4bb4398d20});
BUF_5b4bb4398d20[buf_idx_5b4bb4398d20 * 3 + 0] = tid;
BUF_5b4bb4398d20[buf_idx_5b4bb4398d20 * 3 + 1] = BUF_5b4bb4398620[slot_second5b4bb4398620 * 2 + 0];
BUF_5b4bb4398d20[buf_idx_5b4bb4398d20 * 3 + 2] = BUF_5b4bb4398620[slot_second5b4bb4398620 * 2 + 1];
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_5b4bb429c2c0(uint64_t* BUF_5b4bb439aa50, uint64_t* COUNT5b4bb439b720, HASHTABLE_PROBE HT_5b4bb439aa50, DBI32Type* supplier__s_nationkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_5b4bb439aa50 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_5b4bb439aa50 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_5b4bb439aa50.for_each(KEY_5b4bb439aa50, [&] __device__ (auto const SLOT_5b4bb439aa50) {

auto const [slot_first5b4bb439aa50, slot_second5b4bb439aa50] = SLOT_5b4bb439aa50;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5b4bb439b720, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5b4bb429c2c0(uint64_t* BUF_5b4bb439aa50, uint64_t* BUF_5b4bb439b720, uint64_t* BUF_IDX_5b4bb439b720, HASHTABLE_PROBE HT_5b4bb439aa50, HASHTABLE_INSERT HT_5b4bb439b720, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_5b4bb439aa50 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_5b4bb439aa50 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_5b4bb439aa50.for_each(KEY_5b4bb439aa50, [&] __device__ (auto const SLOT_5b4bb439aa50) {
auto const [slot_first5b4bb439aa50, slot_second5b4bb439aa50] = SLOT_5b4bb439aa50;
if (!(true)) return;
uint64_t KEY_5b4bb439b720 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5b4bb439b720 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5b4bb439b720 = atomicAdd((int*)BUF_IDX_5b4bb439b720, 1);
HT_5b4bb439b720.insert(cuco::pair{KEY_5b4bb439b720, buf_idx_5b4bb439b720});
BUF_5b4bb439b720[buf_idx_5b4bb439b720 * 2 + 0] = tid;
BUF_5b4bb439b720[buf_idx_5b4bb439b720 * 2 + 1] = BUF_5b4bb439aa50[slot_second5b4bb439aa50 * 1 + 0];
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5b4bb431f8f0(uint64_t* BUF_5b4bb4398d20, uint64_t* BUF_5b4bb439b720, HASHTABLE_INSERT HT_5b4bb43527f0, HASHTABLE_PROBE HT_5b4bb4398d20, HASHTABLE_PROBE HT_5b4bb439b720, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n1___n_name, DBStringType* n2___n_name) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9861, Predicate::lte))) return;
uint64_t KEY_5b4bb4398d20 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5b4bb4398d20 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5b4bb4398d20.for_each(KEY_5b4bb4398d20, [&] __device__ (auto const SLOT_5b4bb4398d20) {

auto const [slot_first5b4bb4398d20, slot_second5b4bb4398d20] = SLOT_5b4bb4398d20;
if (!(true)) return;
uint64_t KEY_5b4bb439b720 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_5b4bb439b720 |= reg_lineitem__l_suppkey;
//Probe Hash table
HT_5b4bb439b720.for_each(KEY_5b4bb439b720, [&] __device__ (auto const SLOT_5b4bb439b720) {

auto const [slot_first5b4bb439b720, slot_second5b4bb439b720] = SLOT_5b4bb439b720;
auto reg_n1___n_name = n1___n_name[BUF_5b4bb439b720[slot_second5b4bb439b720 * 2 + 1]];
auto reg_n2___n_name = n2___n_name[BUF_5b4bb4398d20[slot_second5b4bb4398d20 * 3 + 1]];
if (!((((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq))) || ((evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) && (true))) return;
uint64_t KEY_5b4bb43527f0 = 0;
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_lineitem__l_shipdate);

KEY_5b4bb43527f0 |= reg_map0__tmp_attr0;
//Create aggregation hash table
HT_5b4bb43527f0.insert(cuco::pair{KEY_5b4bb43527f0, 1});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5b4bb431f8f0(uint64_t* BUF_5b4bb4398d20, uint64_t* BUF_5b4bb439b720, HASHTABLE_FIND HT_5b4bb43527f0, HASHTABLE_PROBE HT_5b4bb4398d20, HASHTABLE_PROBE HT_5b4bb439b720, DBI64Type* KEY_5b4bb43527f0map0__tmp_attr0, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n1___n_name, DBStringType* n2___n_name) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9861, Predicate::lte))) return;
uint64_t KEY_5b4bb4398d20 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5b4bb4398d20 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5b4bb4398d20.for_each(KEY_5b4bb4398d20, [&] __device__ (auto const SLOT_5b4bb4398d20) {
auto const [slot_first5b4bb4398d20, slot_second5b4bb4398d20] = SLOT_5b4bb4398d20;
if (!(true)) return;
uint64_t KEY_5b4bb439b720 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_5b4bb439b720 |= reg_lineitem__l_suppkey;
//Probe Hash table
HT_5b4bb439b720.for_each(KEY_5b4bb439b720, [&] __device__ (auto const SLOT_5b4bb439b720) {
auto const [slot_first5b4bb439b720, slot_second5b4bb439b720] = SLOT_5b4bb439b720;
auto reg_n1___n_name = n1___n_name[BUF_5b4bb439b720[slot_second5b4bb439b720 * 2 + 1]];
auto reg_n2___n_name = n2___n_name[BUF_5b4bb4398d20[slot_second5b4bb4398d20 * 3 + 1]];
if (!((((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq))) || ((evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) && (true))) return;
uint64_t KEY_5b4bb43527f0 = 0;
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_lineitem__l_shipdate);

KEY_5b4bb43527f0 |= reg_map0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_5b4bb43527f0 = HT_5b4bb43527f0.find(KEY_5b4bb43527f0)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_5b4bb43527f0], reg_map0__tmp_attr1);
KEY_5b4bb43527f0map0__tmp_attr0[buf_idx_5b4bb43527f0] = reg_map0__tmp_attr0;
});
});
}
__global__ void count_5b4bb43bbcf0(size_t COUNT5b4bb43527f0, uint64_t* COUNT5b4bb4365600) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5b4bb43527f0) return;
//Materialize count
atomicAdd((int*)COUNT5b4bb4365600, 1);
}
__global__ void main_5b4bb43bbcf0(size_t COUNT5b4bb43527f0, DBDecimalType* MAT5b4bb4365600aggr0__tmp_attr2, DBI64Type* MAT5b4bb4365600map0__tmp_attr0, uint64_t* MAT_IDX5b4bb4365600, DBDecimalType* aggr0__tmp_attr2, DBI64Type* map0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5b4bb43527f0) return;
//Materialize buffers
auto mat_idx5b4bb4365600 = atomicAdd((int*)MAT_IDX5b4bb4365600, 1);
auto reg_map0__tmp_attr0 = map0__tmp_attr0[tid];
MAT5b4bb4365600map0__tmp_attr0[mat_idx5b4bb4365600] = reg_map0__tmp_attr0;
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT5b4bb4365600aggr0__tmp_attr2[mat_idx5b4bb4365600] = reg_aggr0__tmp_attr2;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT5b4bb439aa50;
hipMalloc(&d_COUNT5b4bb439aa50, sizeof(uint64_t));
hipMemset(d_COUNT5b4bb439aa50, 0, sizeof(uint64_t));
count_5b4bb43a76a0<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT5b4bb439aa50, d_nation__n_name, nation_size);
uint64_t COUNT5b4bb439aa50;
hipMemcpy(&COUNT5b4bb439aa50, d_COUNT5b4bb439aa50, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5b4bb439aa50);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b4bb439aa50;
hipMalloc(&d_BUF_IDX_5b4bb439aa50, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b4bb439aa50, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b4bb439aa50;
hipMalloc(&d_BUF_5b4bb439aa50, sizeof(uint64_t) * COUNT5b4bb439aa50 * 1);
auto d_HT_5b4bb439aa50 = cuco::experimental::static_multimap{ (int)COUNT5b4bb439aa50*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b4bb43a76a0<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_5b4bb439aa50, d_BUF_IDX_5b4bb439aa50, d_HT_5b4bb439aa50.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
hipFree(d_BUF_IDX_5b4bb439aa50);
//Materialize count
uint64_t* d_COUNT5b4bb4399bf0;
hipMalloc(&d_COUNT5b4bb4399bf0, sizeof(uint64_t));
hipMemset(d_COUNT5b4bb4399bf0, 0, sizeof(uint64_t));
count_5b4bb43aa820<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT5b4bb4399bf0, d_nation__n_name, nation_size);
uint64_t COUNT5b4bb4399bf0;
hipMemcpy(&COUNT5b4bb4399bf0, d_COUNT5b4bb4399bf0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5b4bb4399bf0);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b4bb4399bf0;
hipMalloc(&d_BUF_IDX_5b4bb4399bf0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b4bb4399bf0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b4bb4399bf0;
hipMalloc(&d_BUF_5b4bb4399bf0, sizeof(uint64_t) * COUNT5b4bb4399bf0 * 1);
auto d_HT_5b4bb4399bf0 = cuco::experimental::static_multimap{ (int)COUNT5b4bb4399bf0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b4bb43aa820<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_5b4bb4399bf0, d_BUF_IDX_5b4bb4399bf0, d_HT_5b4bb4399bf0.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
hipFree(d_BUF_IDX_5b4bb4399bf0);
//Materialize count
uint64_t* d_COUNT5b4bb4398620;
hipMalloc(&d_COUNT5b4bb4398620, sizeof(uint64_t));
hipMemset(d_COUNT5b4bb4398620, 0, sizeof(uint64_t));
count_5b4bb43ab780<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5b4bb4399bf0, d_COUNT5b4bb4398620, d_HT_5b4bb4399bf0.ref(cuco::for_each), d_customer__c_nationkey, customer_size);
uint64_t COUNT5b4bb4398620;
hipMemcpy(&COUNT5b4bb4398620, d_COUNT5b4bb4398620, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5b4bb4398620);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b4bb4398620;
hipMalloc(&d_BUF_IDX_5b4bb4398620, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b4bb4398620, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b4bb4398620;
hipMalloc(&d_BUF_5b4bb4398620, sizeof(uint64_t) * COUNT5b4bb4398620 * 2);
auto d_HT_5b4bb4398620 = cuco::experimental::static_multimap{ (int)COUNT5b4bb4398620*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b4bb43ab780<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5b4bb4398620, d_BUF_5b4bb4399bf0, d_BUF_IDX_5b4bb4398620, d_HT_5b4bb4398620.ref(cuco::insert), d_HT_5b4bb4399bf0.ref(cuco::for_each), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
hipFree(d_BUF_IDX_5b4bb4398620);
//Materialize count
uint64_t* d_COUNT5b4bb4398d20;
hipMalloc(&d_COUNT5b4bb4398d20, sizeof(uint64_t));
hipMemset(d_COUNT5b4bb4398d20, 0, sizeof(uint64_t));
count_5b4bb43a6160<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5b4bb4398620, d_COUNT5b4bb4398d20, d_HT_5b4bb4398620.ref(cuco::for_each), d_orders__o_custkey, orders_size);
uint64_t COUNT5b4bb4398d20;
hipMemcpy(&COUNT5b4bb4398d20, d_COUNT5b4bb4398d20, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5b4bb4398d20);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b4bb4398d20;
hipMalloc(&d_BUF_IDX_5b4bb4398d20, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b4bb4398d20, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b4bb4398d20;
hipMalloc(&d_BUF_5b4bb4398d20, sizeof(uint64_t) * COUNT5b4bb4398d20 * 3);
auto d_HT_5b4bb4398d20 = cuco::experimental::static_multimap{ (int)COUNT5b4bb4398d20*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b4bb43a6160<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5b4bb4398620, d_BUF_5b4bb4398d20, d_BUF_IDX_5b4bb4398d20, d_HT_5b4bb4398620.ref(cuco::for_each), d_HT_5b4bb4398d20.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderkey, orders_size);
hipFree(d_BUF_IDX_5b4bb4398d20);
//Materialize count
uint64_t* d_COUNT5b4bb439b720;
hipMalloc(&d_COUNT5b4bb439b720, sizeof(uint64_t));
hipMemset(d_COUNT5b4bb439b720, 0, sizeof(uint64_t));
count_5b4bb429c2c0<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_5b4bb439aa50, d_COUNT5b4bb439b720, d_HT_5b4bb439aa50.ref(cuco::for_each), d_supplier__s_nationkey, supplier_size);
uint64_t COUNT5b4bb439b720;
hipMemcpy(&COUNT5b4bb439b720, d_COUNT5b4bb439b720, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5b4bb439b720);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b4bb439b720;
hipMalloc(&d_BUF_IDX_5b4bb439b720, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b4bb439b720, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b4bb439b720;
hipMalloc(&d_BUF_5b4bb439b720, sizeof(uint64_t) * COUNT5b4bb439b720 * 2);
auto d_HT_5b4bb439b720 = cuco::experimental::static_multimap{ (int)COUNT5b4bb439b720*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b4bb429c2c0<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_5b4bb439aa50, d_BUF_5b4bb439b720, d_BUF_IDX_5b4bb439b720, d_HT_5b4bb439aa50.ref(cuco::for_each), d_HT_5b4bb439b720.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
hipFree(d_BUF_IDX_5b4bb439b720);
//Create aggregation hash table
auto d_HT_5b4bb43527f0 = cuco::static_map{ (int)13634*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5b4bb431f8f0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5b4bb4398d20, d_BUF_5b4bb439b720, d_HT_5b4bb43527f0.ref(cuco::insert), d_HT_5b4bb4398d20.ref(cuco::for_each), d_HT_5b4bb439b720.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_nation__n_name);
size_t COUNT5b4bb43527f0 = d_HT_5b4bb43527f0.size();
thrust::device_vector<int64_t> keys_5b4bb43527f0(COUNT5b4bb43527f0), vals_5b4bb43527f0(COUNT5b4bb43527f0);
d_HT_5b4bb43527f0.retrieve_all(keys_5b4bb43527f0.begin(), vals_5b4bb43527f0.begin());
d_HT_5b4bb43527f0.clear();
int64_t* raw_keys5b4bb43527f0 = thrust::raw_pointer_cast(keys_5b4bb43527f0.data());
insertKeys<<<std::ceil((float)COUNT5b4bb43527f0/32.), 32>>>(raw_keys5b4bb43527f0, d_HT_5b4bb43527f0.ref(cuco::insert), COUNT5b4bb43527f0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT5b4bb43527f0);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT5b4bb43527f0);
DBI64Type* d_KEY_5b4bb43527f0map0__tmp_attr0;
hipMalloc(&d_KEY_5b4bb43527f0map0__tmp_attr0, sizeof(DBI64Type) * COUNT5b4bb43527f0);
hipMemset(d_KEY_5b4bb43527f0map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT5b4bb43527f0);
main_5b4bb431f8f0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5b4bb4398d20, d_BUF_5b4bb439b720, d_HT_5b4bb43527f0.ref(cuco::find), d_HT_5b4bb4398d20.ref(cuco::for_each), d_HT_5b4bb439b720.ref(cuco::for_each), d_KEY_5b4bb43527f0map0__tmp_attr0, d_aggr0__tmp_attr2, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_nation__n_name);
//Materialize count
uint64_t* d_COUNT5b4bb4365600;
hipMalloc(&d_COUNT5b4bb4365600, sizeof(uint64_t));
hipMemset(d_COUNT5b4bb4365600, 0, sizeof(uint64_t));
count_5b4bb43bbcf0<<<std::ceil((float)COUNT5b4bb43527f0/32.), 32>>>(COUNT5b4bb43527f0, d_COUNT5b4bb4365600);
uint64_t COUNT5b4bb4365600;
hipMemcpy(&COUNT5b4bb4365600, d_COUNT5b4bb4365600, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5b4bb4365600);
//Materialize buffers
uint64_t* d_MAT_IDX5b4bb4365600;
hipMalloc(&d_MAT_IDX5b4bb4365600, sizeof(uint64_t));
hipMemset(d_MAT_IDX5b4bb4365600, 0, sizeof(uint64_t));
auto MAT5b4bb4365600map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT5b4bb4365600);
DBI64Type* d_MAT5b4bb4365600map0__tmp_attr0;
hipMalloc(&d_MAT5b4bb4365600map0__tmp_attr0, sizeof(DBI64Type) * COUNT5b4bb4365600);
auto MAT5b4bb4365600aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5b4bb4365600);
DBDecimalType* d_MAT5b4bb4365600aggr0__tmp_attr2;
hipMalloc(&d_MAT5b4bb4365600aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT5b4bb4365600);
main_5b4bb43bbcf0<<<std::ceil((float)COUNT5b4bb43527f0/32.), 32>>>(COUNT5b4bb43527f0, d_MAT5b4bb4365600aggr0__tmp_attr2, d_MAT5b4bb4365600map0__tmp_attr0, d_MAT_IDX5b4bb4365600, d_aggr0__tmp_attr2, d_KEY_5b4bb43527f0map0__tmp_attr0);
hipFree(d_MAT_IDX5b4bb4365600);
hipMemcpy(MAT5b4bb4365600map0__tmp_attr0, d_MAT5b4bb4365600map0__tmp_attr0, sizeof(DBI64Type) * COUNT5b4bb4365600, hipMemcpyDeviceToHost);
hipMemcpy(MAT5b4bb4365600aggr0__tmp_attr2, d_MAT5b4bb4365600aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT5b4bb4365600, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5b4bb4365600; i++) { std::cout << MAT5b4bb4365600map0__tmp_attr0[i] << "\t";
std::cout << MAT5b4bb4365600aggr0__tmp_attr2[i] << "\t";
std::cout << std::endl; }
}