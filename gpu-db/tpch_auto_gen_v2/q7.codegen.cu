#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_593adbb43030(uint64_t* COUNT593adbb34a00, DBStringType* n1___n_name, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n1___n_name = n1___n_name[tid];
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
//Materialize count
atomicAdd((int*)COUNT593adbb34a00, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_593adbb43030(uint64_t* BUF_593adbb34a00, uint64_t* BUF_IDX_593adbb34a00, HASHTABLE_INSERT HT_593adbb34a00, DBStringType* n1___n_name, DBI32Type* n1___n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n1___n_name = n1___n_name[tid];
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) || (evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)))) return;
uint64_t KEY_593adbb34a00 = 0;
auto reg_n1___n_nationkey = n1___n_nationkey[tid];

KEY_593adbb34a00 |= reg_n1___n_nationkey;
// Insert hash table kernel;
auto buf_idx_593adbb34a00 = atomicAdd((int*)BUF_IDX_593adbb34a00, 1);
HT_593adbb34a00.insert(cuco::pair{KEY_593adbb34a00, buf_idx_593adbb34a00});
BUF_593adbb34a00[buf_idx_593adbb34a00 * 1 + 0] = tid;
}
__global__ void count_593adbb46470(uint64_t* COUNT593adbb35d50, DBStringType* n2___n_name, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n2___n_name = n2___n_name[tid];
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
//Materialize count
atomicAdd((int*)COUNT593adbb35d50, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_593adbb46470(uint64_t* BUF_593adbb35d50, uint64_t* BUF_IDX_593adbb35d50, HASHTABLE_INSERT HT_593adbb35d50, DBStringType* n2___n_name, DBI32Type* n2___n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
auto reg_n2___n_name = n2___n_name[tid];
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
if (!((evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq)) || (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) return;
uint64_t KEY_593adbb35d50 = 0;
auto reg_n2___n_nationkey = n2___n_nationkey[tid];

KEY_593adbb35d50 |= reg_n2___n_nationkey;
// Insert hash table kernel;
auto buf_idx_593adbb35d50 = atomicAdd((int*)BUF_IDX_593adbb35d50, 1);
HT_593adbb35d50.insert(cuco::pair{KEY_593adbb35d50, buf_idx_593adbb35d50});
BUF_593adbb35d50[buf_idx_593adbb35d50 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_593adbb47430(uint64_t* BUF_593adbb35d50, uint64_t* COUNT593adbb34fe0, HASHTABLE_PROBE HT_593adbb35d50, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_593adbb35d50 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_593adbb35d50 |= reg_customer__c_nationkey;
//Probe Hash table
HT_593adbb35d50.for_each(KEY_593adbb35d50, [&] __device__ (auto const SLOT_593adbb35d50) {

auto const [slot_first593adbb35d50, slot_second593adbb35d50] = SLOT_593adbb35d50;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT593adbb34fe0, 1);
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_593adbb47430(uint64_t* BUF_593adbb34fe0, uint64_t* BUF_593adbb35d50, uint64_t* BUF_IDX_593adbb34fe0, HASHTABLE_INSERT HT_593adbb34fe0, HASHTABLE_PROBE HT_593adbb35d50, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_593adbb35d50 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_593adbb35d50 |= reg_customer__c_nationkey;
//Probe Hash table
HT_593adbb35d50.for_each(KEY_593adbb35d50, [&] __device__ (auto const SLOT_593adbb35d50) {
auto const [slot_first593adbb35d50, slot_second593adbb35d50] = SLOT_593adbb35d50;
if (!(true)) return;
uint64_t KEY_593adbb34fe0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_593adbb34fe0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_593adbb34fe0 = atomicAdd((int*)BUF_IDX_593adbb34fe0, 1);
HT_593adbb34fe0.insert(cuco::pair{KEY_593adbb34fe0, buf_idx_593adbb34fe0});
BUF_593adbb34fe0[buf_idx_593adbb34fe0 * 2 + 0] = BUF_593adbb35d50[slot_second593adbb35d50 * 1 + 0];
BUF_593adbb34fe0[buf_idx_593adbb34fe0 * 2 + 1] = tid;
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_593adbb419a0(uint64_t* BUF_593adbb34fe0, uint64_t* COUNT593adbb36fd0, HASHTABLE_PROBE HT_593adbb34fe0, DBI32Type* orders__o_custkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_593adbb34fe0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_593adbb34fe0 |= reg_orders__o_custkey;
//Probe Hash table
HT_593adbb34fe0.for_each(KEY_593adbb34fe0, [&] __device__ (auto const SLOT_593adbb34fe0) {

auto const [slot_first593adbb34fe0, slot_second593adbb34fe0] = SLOT_593adbb34fe0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT593adbb36fd0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_593adbb419a0(uint64_t* BUF_593adbb34fe0, uint64_t* BUF_593adbb36fd0, uint64_t* BUF_IDX_593adbb36fd0, HASHTABLE_PROBE HT_593adbb34fe0, HASHTABLE_INSERT HT_593adbb36fd0, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_593adbb34fe0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_593adbb34fe0 |= reg_orders__o_custkey;
//Probe Hash table
HT_593adbb34fe0.for_each(KEY_593adbb34fe0, [&] __device__ (auto const SLOT_593adbb34fe0) {
auto const [slot_first593adbb34fe0, slot_second593adbb34fe0] = SLOT_593adbb34fe0;
if (!(true)) return;
uint64_t KEY_593adbb36fd0 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_593adbb36fd0 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_593adbb36fd0 = atomicAdd((int*)BUF_IDX_593adbb36fd0, 1);
HT_593adbb36fd0.insert(cuco::pair{KEY_593adbb36fd0, buf_idx_593adbb36fd0});
BUF_593adbb36fd0[buf_idx_593adbb36fd0 * 3 + 0] = tid;
BUF_593adbb36fd0[buf_idx_593adbb36fd0 * 3 + 1] = BUF_593adbb34fe0[slot_second593adbb34fe0 * 2 + 0];
BUF_593adbb36fd0[buf_idx_593adbb36fd0 * 3 + 2] = BUF_593adbb34fe0[slot_second593adbb34fe0 * 2 + 1];
});
}
template<typename HASHTABLE_PROBE>
__global__ void count_593adbb1fa30(uint64_t* BUF_593adbb34a00, uint64_t* COUNT593adbb370e0, HASHTABLE_PROBE HT_593adbb34a00, DBI32Type* supplier__s_nationkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_593adbb34a00 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_593adbb34a00 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_593adbb34a00.for_each(KEY_593adbb34a00, [&] __device__ (auto const SLOT_593adbb34a00) {

auto const [slot_first593adbb34a00, slot_second593adbb34a00] = SLOT_593adbb34a00;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT593adbb370e0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_593adbb1fa30(uint64_t* BUF_593adbb34a00, uint64_t* BUF_593adbb370e0, uint64_t* BUF_IDX_593adbb370e0, HASHTABLE_PROBE HT_593adbb34a00, HASHTABLE_INSERT HT_593adbb370e0, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_593adbb34a00 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_593adbb34a00 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_593adbb34a00.for_each(KEY_593adbb34a00, [&] __device__ (auto const SLOT_593adbb34a00) {
auto const [slot_first593adbb34a00, slot_second593adbb34a00] = SLOT_593adbb34a00;
if (!(true)) return;
uint64_t KEY_593adbb370e0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_593adbb370e0 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_593adbb370e0 = atomicAdd((int*)BUF_IDX_593adbb370e0, 1);
HT_593adbb370e0.insert(cuco::pair{KEY_593adbb370e0, buf_idx_593adbb370e0});
BUF_593adbb370e0[buf_idx_593adbb370e0 * 2 + 0] = tid;
BUF_593adbb370e0[buf_idx_593adbb370e0 * 2 + 1] = BUF_593adbb34a00[slot_second593adbb34a00 * 1 + 0];
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_593adbb20000(uint64_t* BUF_593adbb36fd0, uint64_t* BUF_593adbb370e0, HASHTABLE_INSERT HT_593adbaee6e0, HASHTABLE_PROBE HT_593adbb36fd0, HASHTABLE_PROBE HT_593adbb370e0, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n1___n_name, DBStringType* n2___n_name) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9861, Predicate::lte))) return;
uint64_t KEY_593adbb36fd0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_593adbb36fd0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_593adbb36fd0.for_each(KEY_593adbb36fd0, [&] __device__ (auto const SLOT_593adbb36fd0) {

auto const [slot_first593adbb36fd0, slot_second593adbb36fd0] = SLOT_593adbb36fd0;
if (!(true)) return;
uint64_t KEY_593adbb370e0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_593adbb370e0 |= reg_lineitem__l_suppkey;
//Probe Hash table
HT_593adbb370e0.for_each(KEY_593adbb370e0, [&] __device__ (auto const SLOT_593adbb370e0) {

auto const [slot_first593adbb370e0, slot_second593adbb370e0] = SLOT_593adbb370e0;
auto reg_n1___n_name = n1___n_name[BUF_593adbb370e0[slot_second593adbb370e0 * 2 + 1]];
auto reg_n2___n_name = n2___n_name[BUF_593adbb36fd0[slot_second593adbb36fd0 * 3 + 1]];
if (!((((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq))) || ((evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) && (true))) return;
uint64_t KEY_593adbaee6e0 = 0;
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_lineitem__l_shipdate);

KEY_593adbaee6e0 |= reg_map0__tmp_attr0;
//Create aggregation hash table
HT_593adbaee6e0.insert(cuco::pair{KEY_593adbaee6e0, 1});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_593adbb20000(uint64_t* BUF_593adbb36fd0, uint64_t* BUF_593adbb370e0, HASHTABLE_FIND HT_593adbaee6e0, HASHTABLE_PROBE HT_593adbb36fd0, HASHTABLE_PROBE HT_593adbb370e0, DBI64Type* KEY_593adbaee6e0map0__tmp_attr0, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBStringType* n1___n_name, DBStringType* n2___n_name) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9861, Predicate::lte))) return;
uint64_t KEY_593adbb36fd0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_593adbb36fd0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_593adbb36fd0.for_each(KEY_593adbb36fd0, [&] __device__ (auto const SLOT_593adbb36fd0) {
auto const [slot_first593adbb36fd0, slot_second593adbb36fd0] = SLOT_593adbb36fd0;
if (!(true)) return;
uint64_t KEY_593adbb370e0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_593adbb370e0 |= reg_lineitem__l_suppkey;
//Probe Hash table
HT_593adbb370e0.for_each(KEY_593adbb370e0, [&] __device__ (auto const SLOT_593adbb370e0) {
auto const [slot_first593adbb370e0, slot_second593adbb370e0] = SLOT_593adbb370e0;
auto reg_n1___n_name = n1___n_name[BUF_593adbb370e0[slot_second593adbb370e0 * 2 + 1]];
auto reg_n2___n_name = n2___n_name[BUF_593adbb36fd0[slot_second593adbb36fd0 * 3 + 1]];
if (!((((evaluatePredicate(reg_n1___n_name, "FRANCE", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "GERMANY", Predicate::eq))) || ((evaluatePredicate(reg_n1___n_name, "GERMANY", Predicate::eq)) && (evaluatePredicate(reg_n2___n_name, "FRANCE", Predicate::eq)))) && (true))) return;
uint64_t KEY_593adbaee6e0 = 0;
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_lineitem__l_shipdate);

KEY_593adbaee6e0 |= reg_map0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_593adbaee6e0 = HT_593adbaee6e0.find(KEY_593adbaee6e0)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_593adbaee6e0], reg_map0__tmp_attr1);
KEY_593adbaee6e0map0__tmp_attr0[buf_idx_593adbaee6e0] = reg_map0__tmp_attr0;
});
});
}
__global__ void count_593adbb57100(size_t COUNT593adbaee6e0, uint64_t* COUNT593adbb01dc0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT593adbaee6e0) return;
//Materialize count
atomicAdd((int*)COUNT593adbb01dc0, 1);
}
__global__ void main_593adbb57100(size_t COUNT593adbaee6e0, DBDecimalType* MAT593adbb01dc0aggr0__tmp_attr2, DBI64Type* MAT593adbb01dc0map0__tmp_attr0, uint64_t* MAT_IDX593adbb01dc0, DBDecimalType* aggr0__tmp_attr2, DBI64Type* map0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT593adbaee6e0) return;
//Materialize buffers
auto mat_idx593adbb01dc0 = atomicAdd((int*)MAT_IDX593adbb01dc0, 1);
auto reg_map0__tmp_attr0 = map0__tmp_attr0[tid];
MAT593adbb01dc0map0__tmp_attr0[mat_idx593adbb01dc0] = reg_map0__tmp_attr0;
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT593adbb01dc0aggr0__tmp_attr2[mat_idx593adbb01dc0] = reg_aggr0__tmp_attr2;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT593adbb34a00;
hipMalloc(&d_COUNT593adbb34a00, sizeof(uint64_t));
hipMemset(d_COUNT593adbb34a00, 0, sizeof(uint64_t));
count_593adbb43030<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT593adbb34a00, d_nation__n_name, nation_size);
uint64_t COUNT593adbb34a00;
hipMemcpy(&COUNT593adbb34a00, d_COUNT593adbb34a00, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_593adbb34a00;
hipMalloc(&d_BUF_IDX_593adbb34a00, sizeof(uint64_t));
hipMemset(d_BUF_IDX_593adbb34a00, 0, sizeof(uint64_t));
uint64_t* d_BUF_593adbb34a00;
hipMalloc(&d_BUF_593adbb34a00, sizeof(uint64_t) * COUNT593adbb34a00 * 1);
auto d_HT_593adbb34a00 = cuco::experimental::static_multimap{ (int)COUNT593adbb34a00*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_593adbb43030<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_593adbb34a00, d_BUF_IDX_593adbb34a00, d_HT_593adbb34a00.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
//Materialize count
uint64_t* d_COUNT593adbb35d50;
hipMalloc(&d_COUNT593adbb35d50, sizeof(uint64_t));
hipMemset(d_COUNT593adbb35d50, 0, sizeof(uint64_t));
count_593adbb46470<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT593adbb35d50, d_nation__n_name, nation_size);
uint64_t COUNT593adbb35d50;
hipMemcpy(&COUNT593adbb35d50, d_COUNT593adbb35d50, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_593adbb35d50;
hipMalloc(&d_BUF_IDX_593adbb35d50, sizeof(uint64_t));
hipMemset(d_BUF_IDX_593adbb35d50, 0, sizeof(uint64_t));
uint64_t* d_BUF_593adbb35d50;
hipMalloc(&d_BUF_593adbb35d50, sizeof(uint64_t) * COUNT593adbb35d50 * 1);
auto d_HT_593adbb35d50 = cuco::experimental::static_multimap{ (int)COUNT593adbb35d50*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_593adbb46470<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_593adbb35d50, d_BUF_IDX_593adbb35d50, d_HT_593adbb35d50.ref(cuco::insert), d_nation__n_name, d_nation__n_nationkey, nation_size);
//Materialize count
uint64_t* d_COUNT593adbb34fe0;
hipMalloc(&d_COUNT593adbb34fe0, sizeof(uint64_t));
hipMemset(d_COUNT593adbb34fe0, 0, sizeof(uint64_t));
count_593adbb47430<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_593adbb35d50, d_COUNT593adbb34fe0, d_HT_593adbb35d50.ref(cuco::for_each), d_customer__c_nationkey, customer_size);
uint64_t COUNT593adbb34fe0;
hipMemcpy(&COUNT593adbb34fe0, d_COUNT593adbb34fe0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_593adbb34fe0;
hipMalloc(&d_BUF_IDX_593adbb34fe0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_593adbb34fe0, 0, sizeof(uint64_t));
uint64_t* d_BUF_593adbb34fe0;
hipMalloc(&d_BUF_593adbb34fe0, sizeof(uint64_t) * COUNT593adbb34fe0 * 2);
auto d_HT_593adbb34fe0 = cuco::experimental::static_multimap{ (int)COUNT593adbb34fe0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_593adbb47430<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_593adbb34fe0, d_BUF_593adbb35d50, d_BUF_IDX_593adbb34fe0, d_HT_593adbb34fe0.ref(cuco::insert), d_HT_593adbb35d50.ref(cuco::for_each), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
//Materialize count
uint64_t* d_COUNT593adbb36fd0;
hipMalloc(&d_COUNT593adbb36fd0, sizeof(uint64_t));
hipMemset(d_COUNT593adbb36fd0, 0, sizeof(uint64_t));
count_593adbb419a0<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_593adbb34fe0, d_COUNT593adbb36fd0, d_HT_593adbb34fe0.ref(cuco::for_each), d_orders__o_custkey, orders_size);
uint64_t COUNT593adbb36fd0;
hipMemcpy(&COUNT593adbb36fd0, d_COUNT593adbb36fd0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_593adbb36fd0;
hipMalloc(&d_BUF_IDX_593adbb36fd0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_593adbb36fd0, 0, sizeof(uint64_t));
uint64_t* d_BUF_593adbb36fd0;
hipMalloc(&d_BUF_593adbb36fd0, sizeof(uint64_t) * COUNT593adbb36fd0 * 3);
auto d_HT_593adbb36fd0 = cuco::experimental::static_multimap{ (int)COUNT593adbb36fd0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_593adbb419a0<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_593adbb34fe0, d_BUF_593adbb36fd0, d_BUF_IDX_593adbb36fd0, d_HT_593adbb34fe0.ref(cuco::for_each), d_HT_593adbb36fd0.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderkey, orders_size);
//Materialize count
uint64_t* d_COUNT593adbb370e0;
hipMalloc(&d_COUNT593adbb370e0, sizeof(uint64_t));
hipMemset(d_COUNT593adbb370e0, 0, sizeof(uint64_t));
count_593adbb1fa30<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_593adbb34a00, d_COUNT593adbb370e0, d_HT_593adbb34a00.ref(cuco::for_each), d_supplier__s_nationkey, supplier_size);
uint64_t COUNT593adbb370e0;
hipMemcpy(&COUNT593adbb370e0, d_COUNT593adbb370e0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_593adbb370e0;
hipMalloc(&d_BUF_IDX_593adbb370e0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_593adbb370e0, 0, sizeof(uint64_t));
uint64_t* d_BUF_593adbb370e0;
hipMalloc(&d_BUF_593adbb370e0, sizeof(uint64_t) * COUNT593adbb370e0 * 2);
auto d_HT_593adbb370e0 = cuco::experimental::static_multimap{ (int)COUNT593adbb370e0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_593adbb1fa30<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_593adbb34a00, d_BUF_593adbb370e0, d_BUF_IDX_593adbb370e0, d_HT_593adbb34a00.ref(cuco::for_each), d_HT_593adbb370e0.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
//Create aggregation hash table
auto d_HT_593adbaee6e0 = cuco::static_map{ (int)13634*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_593adbb20000<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_593adbb36fd0, d_BUF_593adbb370e0, d_HT_593adbaee6e0.ref(cuco::insert), d_HT_593adbb36fd0.ref(cuco::for_each), d_HT_593adbb370e0.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_nation__n_name);
size_t COUNT593adbaee6e0 = d_HT_593adbaee6e0.size();
thrust::device_vector<int64_t> keys_593adbaee6e0(COUNT593adbaee6e0), vals_593adbaee6e0(COUNT593adbaee6e0);
d_HT_593adbaee6e0.retrieve_all(keys_593adbaee6e0.begin(), vals_593adbaee6e0.begin());
d_HT_593adbaee6e0.clear();
int64_t* raw_keys593adbaee6e0 = thrust::raw_pointer_cast(keys_593adbaee6e0.data());
insertKeys<<<std::ceil((float)COUNT593adbaee6e0/32.), 32>>>(raw_keys593adbaee6e0, d_HT_593adbaee6e0.ref(cuco::insert), COUNT593adbaee6e0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT593adbaee6e0);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT593adbaee6e0);
DBI64Type* d_KEY_593adbaee6e0map0__tmp_attr0;
hipMalloc(&d_KEY_593adbaee6e0map0__tmp_attr0, sizeof(DBI64Type) * COUNT593adbaee6e0);
hipMemset(d_KEY_593adbaee6e0map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT593adbaee6e0);
main_593adbb20000<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_593adbb36fd0, d_BUF_593adbb370e0, d_HT_593adbaee6e0.ref(cuco::find), d_HT_593adbb36fd0.ref(cuco::for_each), d_HT_593adbb370e0.ref(cuco::for_each), d_KEY_593adbaee6e0map0__tmp_attr0, d_aggr0__tmp_attr2, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, d_lineitem__l_suppkey, lineitem_size, d_nation__n_name, d_nation__n_name);
//Materialize count
uint64_t* d_COUNT593adbb01dc0;
hipMalloc(&d_COUNT593adbb01dc0, sizeof(uint64_t));
hipMemset(d_COUNT593adbb01dc0, 0, sizeof(uint64_t));
count_593adbb57100<<<std::ceil((float)COUNT593adbaee6e0/32.), 32>>>(COUNT593adbaee6e0, d_COUNT593adbb01dc0);
uint64_t COUNT593adbb01dc0;
hipMemcpy(&COUNT593adbb01dc0, d_COUNT593adbb01dc0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX593adbb01dc0;
hipMalloc(&d_MAT_IDX593adbb01dc0, sizeof(uint64_t));
hipMemset(d_MAT_IDX593adbb01dc0, 0, sizeof(uint64_t));
auto MAT593adbb01dc0map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT593adbb01dc0);
DBI64Type* d_MAT593adbb01dc0map0__tmp_attr0;
hipMalloc(&d_MAT593adbb01dc0map0__tmp_attr0, sizeof(DBI64Type) * COUNT593adbb01dc0);
auto MAT593adbb01dc0aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT593adbb01dc0);
DBDecimalType* d_MAT593adbb01dc0aggr0__tmp_attr2;
hipMalloc(&d_MAT593adbb01dc0aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT593adbb01dc0);
main_593adbb57100<<<std::ceil((float)COUNT593adbaee6e0/32.), 32>>>(COUNT593adbaee6e0, d_MAT593adbb01dc0aggr0__tmp_attr2, d_MAT593adbb01dc0map0__tmp_attr0, d_MAT_IDX593adbb01dc0, d_aggr0__tmp_attr2, d_KEY_593adbaee6e0map0__tmp_attr0);
hipMemcpy(MAT593adbb01dc0map0__tmp_attr0, d_MAT593adbb01dc0map0__tmp_attr0, sizeof(DBI64Type) * COUNT593adbb01dc0, hipMemcpyDeviceToHost);
hipMemcpy(MAT593adbb01dc0aggr0__tmp_attr2, d_MAT593adbb01dc0aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT593adbb01dc0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT593adbb01dc0; i++) { std::cout << MAT593adbb01dc0map0__tmp_attr0[i] << "\t";
std::cout << MAT593adbb01dc0aggr0__tmp_attr2[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_593adbb34a00);
hipFree(d_BUF_IDX_593adbb34a00);
hipFree(d_COUNT593adbb34a00);
hipFree(d_BUF_593adbb35d50);
hipFree(d_BUF_IDX_593adbb35d50);
hipFree(d_COUNT593adbb35d50);
hipFree(d_BUF_593adbb34fe0);
hipFree(d_BUF_IDX_593adbb34fe0);
hipFree(d_COUNT593adbb34fe0);
hipFree(d_BUF_593adbb36fd0);
hipFree(d_BUF_IDX_593adbb36fd0);
hipFree(d_COUNT593adbb36fd0);
hipFree(d_BUF_593adbb370e0);
hipFree(d_BUF_IDX_593adbb370e0);
hipFree(d_COUNT593adbb370e0);
hipFree(d_KEY_593adbaee6e0map0__tmp_attr0);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_COUNT593adbb01dc0);
hipFree(d_MAT593adbb01dc0aggr0__tmp_attr2);
hipFree(d_MAT593adbb01dc0map0__tmp_attr0);
hipFree(d_MAT_IDX593adbb01dc0);
free(MAT593adbb01dc0aggr0__tmp_attr2);
free(MAT593adbb01dc0map0__tmp_attr0);
}