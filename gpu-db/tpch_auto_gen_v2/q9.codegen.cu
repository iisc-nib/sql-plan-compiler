#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_615ba43438a0(uint64_t* COUNT615ba4334f00, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
//Materialize count
atomicAdd((int*)COUNT615ba4334f00, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_615ba43438a0(uint64_t* BUF_615ba4334f00, uint64_t* BUF_IDX_615ba4334f00, HASHTABLE_INSERT HT_615ba4334f00, DBI32Type* nation__n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_615ba4334f00 = 0;
auto reg_nation__n_nationkey = nation__n_nationkey[tid];

KEY_615ba4334f00 |= reg_nation__n_nationkey;
// Insert hash table kernel;
auto buf_idx_615ba4334f00 = atomicAdd((int*)BUF_IDX_615ba4334f00, 1);
HT_615ba4334f00.insert(cuco::pair{KEY_615ba4334f00, buf_idx_615ba4334f00});
BUF_615ba4334f00[buf_idx_615ba4334f00 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_615ba42bf660(uint64_t* BUF_615ba4334f00, uint64_t* COUNT615ba4334780, HASHTABLE_PROBE HT_615ba4334f00, DBI32Type* supplier__s_nationkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_615ba4334f00 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_615ba4334f00 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_615ba4334f00.for_each(KEY_615ba4334f00, [&] __device__ (auto const SLOT_615ba4334f00) {

auto const [slot_first615ba4334f00, slot_second615ba4334f00] = SLOT_615ba4334f00;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT615ba4334780, 1);
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void main_615ba42bf660(uint64_t* BUF_615ba4334780, uint64_t* BUF_615ba4334f00, uint64_t* BUF_IDX_615ba4334780, HASHTABLE_INSERT HT_615ba4334780, HASHTABLE_PROBE HT_615ba4334f00, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_615ba4334f00 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_615ba4334f00 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_615ba4334f00.for_each(KEY_615ba4334f00, [&] __device__ (auto const SLOT_615ba4334f00) {
auto const [slot_first615ba4334f00, slot_second615ba4334f00] = SLOT_615ba4334f00;
if (!(true)) return;
uint64_t KEY_615ba4334780 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_615ba4334780 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_615ba4334780 = atomicAdd((int*)BUF_IDX_615ba4334780, 1);
HT_615ba4334780.insert(cuco::pair{KEY_615ba4334780, buf_idx_615ba4334780});
BUF_615ba4334780[buf_idx_615ba4334780 * 2 + 0] = tid;
BUF_615ba4334780[buf_idx_615ba4334780 * 2 + 1] = BUF_615ba4334f00[slot_second615ba4334f00 * 1 + 0];
});
}
__global__ void count_615ba4348d10(uint64_t* COUNT615ba433cfe0, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
//Materialize count
atomicAdd((int*)COUNT615ba433cfe0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_615ba4348d10(uint64_t* BUF_615ba433cfe0, uint64_t* BUF_IDX_615ba433cfe0, HASHTABLE_INSERT HT_615ba433cfe0, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_615ba433cfe0 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_615ba433cfe0 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_615ba433cfe0 = atomicAdd((int*)BUF_IDX_615ba433cfe0, 1);
HT_615ba433cfe0.insert(cuco::pair{KEY_615ba433cfe0, buf_idx_615ba433cfe0});
BUF_615ba433cfe0[buf_idx_615ba433cfe0 * 1 + 0] = tid;
}
__global__ void count_615ba423c2c0(uint64_t* COUNT615ba433d0a0, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
//Materialize count
atomicAdd((int*)COUNT615ba433d0a0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_615ba423c2c0(uint64_t* BUF_615ba433d0a0, uint64_t* BUF_IDX_615ba433d0a0, HASHTABLE_INSERT HT_615ba433d0a0, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
uint64_t KEY_615ba433d0a0 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_615ba433d0a0 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_615ba433d0a0 = atomicAdd((int*)BUF_IDX_615ba433d0a0, 1);
HT_615ba433d0a0.insert(cuco::pair{KEY_615ba433d0a0, buf_idx_615ba433d0a0});
BUF_615ba433d0a0[buf_idx_615ba433d0a0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_615ba43429c0(uint64_t* BUF_615ba4334780, uint64_t* COUNT615ba433d1b0, HASHTABLE_PROBE HT_615ba4334780, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= partsupp_size) return;
uint64_t KEY_615ba4334780 = 0;
auto reg_partsupp__ps_suppkey = partsupp__ps_suppkey[tid];

KEY_615ba4334780 |= reg_partsupp__ps_suppkey;
//Probe Hash table
HT_615ba4334780.for_each(KEY_615ba4334780, [&] __device__ (auto const SLOT_615ba4334780) {

auto const [slot_first615ba4334780, slot_second615ba4334780] = SLOT_615ba4334780;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT615ba433d1b0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_615ba43429c0(uint64_t* BUF_615ba4334780, uint64_t* BUF_615ba433d1b0, uint64_t* BUF_IDX_615ba433d1b0, HASHTABLE_PROBE HT_615ba4334780, HASHTABLE_INSERT HT_615ba433d1b0, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size, DBI32Type* supplier__s_suppkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= partsupp_size) return;
uint64_t KEY_615ba4334780 = 0;
auto reg_partsupp__ps_suppkey = partsupp__ps_suppkey[tid];

KEY_615ba4334780 |= reg_partsupp__ps_suppkey;
//Probe Hash table
HT_615ba4334780.for_each(KEY_615ba4334780, [&] __device__ (auto const SLOT_615ba4334780) {
auto const [slot_first615ba4334780, slot_second615ba4334780] = SLOT_615ba4334780;
if (!(true)) return;
uint64_t KEY_615ba433d1b0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[BUF_615ba4334780[slot_second615ba4334780 * 2 + 0]];

KEY_615ba433d1b0 |= reg_supplier__s_suppkey;
auto reg_partsupp__ps_partkey = partsupp__ps_partkey[tid];
KEY_615ba433d1b0 <<= 32;
KEY_615ba433d1b0 |= reg_partsupp__ps_partkey;
// Insert hash table kernel;
auto buf_idx_615ba433d1b0 = atomicAdd((int*)BUF_IDX_615ba433d1b0, 1);
HT_615ba433d1b0.insert(cuco::pair{KEY_615ba433d1b0, buf_idx_615ba433d1b0});
BUF_615ba433d1b0[buf_idx_615ba433d1b0 * 3 + 0] = BUF_615ba4334780[slot_second615ba4334780 * 2 + 0];
BUF_615ba433d1b0[buf_idx_615ba433d1b0 * 3 + 1] = tid;
BUF_615ba433d1b0[buf_idx_615ba433d1b0 * 3 + 2] = BUF_615ba4334780[slot_second615ba4334780 * 2 + 1];
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_615ba4340570(uint64_t* BUF_615ba433cfe0, uint64_t* BUF_615ba433d0a0, uint64_t* BUF_615ba433d1b0, HASHTABLE_INSERT HT_615ba42f1d30, HASHTABLE_PROBE HT_615ba433cfe0, HASHTABLE_PROBE HT_615ba433d0a0, HASHTABLE_PROBE HT_615ba433d1b0, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBDateType* orders__o_orderdate) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_615ba433cfe0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_615ba433cfe0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_615ba433cfe0.for_each(KEY_615ba433cfe0, [&] __device__ (auto const SLOT_615ba433cfe0) {

auto const [slot_first615ba433cfe0, slot_second615ba433cfe0] = SLOT_615ba433cfe0;
if (!(true)) return;
uint64_t KEY_615ba433d0a0 = 0;
auto reg_lineitem__l_partkey = lineitem__l_partkey[tid];

KEY_615ba433d0a0 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_615ba433d0a0.for_each(KEY_615ba433d0a0, [&] __device__ (auto const SLOT_615ba433d0a0) {

auto const [slot_first615ba433d0a0, slot_second615ba433d0a0] = SLOT_615ba433d0a0;
if (!(true)) return;
uint64_t KEY_615ba433d1b0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_615ba433d1b0 |= reg_lineitem__l_suppkey;
KEY_615ba433d1b0 <<= 32;
KEY_615ba433d1b0 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_615ba433d1b0.for_each(KEY_615ba433d1b0, [&] __device__ (auto const SLOT_615ba433d1b0) {

auto const [slot_first615ba433d1b0, slot_second615ba433d1b0] = SLOT_615ba433d1b0;
if (!(true)) return;
uint64_t KEY_615ba42f1d30 = 0;
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_615ba433cfe0[slot_second615ba433cfe0 * 1 + 0]];
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_orders__o_orderdate);

KEY_615ba42f1d30 |= reg_map0__tmp_attr0;
//Create aggregation hash table
HT_615ba42f1d30.insert(cuco::pair{KEY_615ba42f1d30, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_615ba4340570(uint64_t* BUF_615ba433cfe0, uint64_t* BUF_615ba433d0a0, uint64_t* BUF_615ba433d1b0, HASHTABLE_FIND HT_615ba42f1d30, HASHTABLE_PROBE HT_615ba433cfe0, HASHTABLE_PROBE HT_615ba433d0a0, HASHTABLE_PROBE HT_615ba433d1b0, DBI64Type* KEY_615ba42f1d30map0__tmp_attr0, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBDateType* orders__o_orderdate, DBDecimalType* partsupp__ps_supplycost) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_615ba433cfe0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_615ba433cfe0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_615ba433cfe0.for_each(KEY_615ba433cfe0, [&] __device__ (auto const SLOT_615ba433cfe0) {
auto const [slot_first615ba433cfe0, slot_second615ba433cfe0] = SLOT_615ba433cfe0;
if (!(true)) return;
uint64_t KEY_615ba433d0a0 = 0;
auto reg_lineitem__l_partkey = lineitem__l_partkey[tid];

KEY_615ba433d0a0 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_615ba433d0a0.for_each(KEY_615ba433d0a0, [&] __device__ (auto const SLOT_615ba433d0a0) {
auto const [slot_first615ba433d0a0, slot_second615ba433d0a0] = SLOT_615ba433d0a0;
if (!(true)) return;
uint64_t KEY_615ba433d1b0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_615ba433d1b0 |= reg_lineitem__l_suppkey;
KEY_615ba433d1b0 <<= 32;
KEY_615ba433d1b0 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_615ba433d1b0.for_each(KEY_615ba433d1b0, [&] __device__ (auto const SLOT_615ba433d1b0) {
auto const [slot_first615ba433d1b0, slot_second615ba433d1b0] = SLOT_615ba433d1b0;
if (!(true)) return;
uint64_t KEY_615ba42f1d30 = 0;
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_615ba433cfe0[slot_second615ba433cfe0 * 1 + 0]];
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_orders__o_orderdate);

KEY_615ba42f1d30 |= reg_map0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_615ba42f1d30 = HT_615ba42f1d30.find(KEY_615ba42f1d30)->second;
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
auto reg_partsupp__ps_supplycost = partsupp__ps_supplycost[BUF_615ba433d1b0[slot_second615ba433d1b0 * 3 + 1]];
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = ((reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount))) - ((reg_partsupp__ps_supplycost) * (reg_lineitem__l_quantity));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_615ba42f1d30], reg_map0__tmp_attr1);
KEY_615ba42f1d30map0__tmp_attr0[buf_idx_615ba42f1d30] = reg_map0__tmp_attr0;
});
});
});
}
__global__ void count_615ba4356b90(size_t COUNT615ba42f1d30, uint64_t* COUNT615ba43041f0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT615ba42f1d30) return;
//Materialize count
atomicAdd((int*)COUNT615ba43041f0, 1);
}
__global__ void main_615ba4356b90(size_t COUNT615ba42f1d30, DBDecimalType* MAT615ba43041f0aggr0__tmp_attr2, DBI64Type* MAT615ba43041f0map0__tmp_attr0, uint64_t* MAT_IDX615ba43041f0, DBDecimalType* aggr0__tmp_attr2, DBI64Type* map0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT615ba42f1d30) return;
//Materialize buffers
auto mat_idx615ba43041f0 = atomicAdd((int*)MAT_IDX615ba43041f0, 1);
auto reg_map0__tmp_attr0 = map0__tmp_attr0[tid];
MAT615ba43041f0map0__tmp_attr0[mat_idx615ba43041f0] = reg_map0__tmp_attr0;
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT615ba43041f0aggr0__tmp_attr2[mat_idx615ba43041f0] = reg_aggr0__tmp_attr2;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT615ba4334f00;
hipMalloc(&d_COUNT615ba4334f00, sizeof(uint64_t));
hipMemset(d_COUNT615ba4334f00, 0, sizeof(uint64_t));
count_615ba43438a0<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT615ba4334f00, nation_size);
uint64_t COUNT615ba4334f00;
hipMemcpy(&COUNT615ba4334f00, d_COUNT615ba4334f00, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT615ba4334f00);
// Insert hash table control;
uint64_t* d_BUF_IDX_615ba4334f00;
hipMalloc(&d_BUF_IDX_615ba4334f00, sizeof(uint64_t));
hipMemset(d_BUF_IDX_615ba4334f00, 0, sizeof(uint64_t));
uint64_t* d_BUF_615ba4334f00;
hipMalloc(&d_BUF_615ba4334f00, sizeof(uint64_t) * COUNT615ba4334f00 * 1);
auto d_HT_615ba4334f00 = cuco::experimental::static_multimap{ (int)COUNT615ba4334f00*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_615ba43438a0<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_615ba4334f00, d_BUF_IDX_615ba4334f00, d_HT_615ba4334f00.ref(cuco::insert), d_nation__n_nationkey, nation_size);
hipFree(d_BUF_IDX_615ba4334f00);
//Materialize count
uint64_t* d_COUNT615ba4334780;
hipMalloc(&d_COUNT615ba4334780, sizeof(uint64_t));
hipMemset(d_COUNT615ba4334780, 0, sizeof(uint64_t));
count_615ba42bf660<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_615ba4334f00, d_COUNT615ba4334780, d_HT_615ba4334f00.ref(cuco::for_each), d_supplier__s_nationkey, supplier_size);
uint64_t COUNT615ba4334780;
hipMemcpy(&COUNT615ba4334780, d_COUNT615ba4334780, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT615ba4334780);
// Insert hash table control;
uint64_t* d_BUF_IDX_615ba4334780;
hipMalloc(&d_BUF_IDX_615ba4334780, sizeof(uint64_t));
hipMemset(d_BUF_IDX_615ba4334780, 0, sizeof(uint64_t));
uint64_t* d_BUF_615ba4334780;
hipMalloc(&d_BUF_615ba4334780, sizeof(uint64_t) * COUNT615ba4334780 * 2);
auto d_HT_615ba4334780 = cuco::experimental::static_multimap{ (int)COUNT615ba4334780*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_615ba42bf660<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_615ba4334780, d_BUF_615ba4334f00, d_BUF_IDX_615ba4334780, d_HT_615ba4334780.ref(cuco::insert), d_HT_615ba4334f00.ref(cuco::for_each), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
hipFree(d_BUF_IDX_615ba4334780);
//Materialize count
uint64_t* d_COUNT615ba433cfe0;
hipMalloc(&d_COUNT615ba433cfe0, sizeof(uint64_t));
hipMemset(d_COUNT615ba433cfe0, 0, sizeof(uint64_t));
count_615ba4348d10<<<std::ceil((float)orders_size/32.), 32>>>(d_COUNT615ba433cfe0, orders_size);
uint64_t COUNT615ba433cfe0;
hipMemcpy(&COUNT615ba433cfe0, d_COUNT615ba433cfe0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT615ba433cfe0);
// Insert hash table control;
uint64_t* d_BUF_IDX_615ba433cfe0;
hipMalloc(&d_BUF_IDX_615ba433cfe0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_615ba433cfe0, 0, sizeof(uint64_t));
uint64_t* d_BUF_615ba433cfe0;
hipMalloc(&d_BUF_615ba433cfe0, sizeof(uint64_t) * COUNT615ba433cfe0 * 1);
auto d_HT_615ba433cfe0 = cuco::experimental::static_multimap{ (int)COUNT615ba433cfe0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_615ba4348d10<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_615ba433cfe0, d_BUF_IDX_615ba433cfe0, d_HT_615ba433cfe0.ref(cuco::insert), d_orders__o_orderkey, orders_size);
hipFree(d_BUF_IDX_615ba433cfe0);
//Materialize count
uint64_t* d_COUNT615ba433d0a0;
hipMalloc(&d_COUNT615ba433d0a0, sizeof(uint64_t));
hipMemset(d_COUNT615ba433d0a0, 0, sizeof(uint64_t));
count_615ba423c2c0<<<std::ceil((float)part_size/32.), 32>>>(d_COUNT615ba433d0a0, part_size);
uint64_t COUNT615ba433d0a0;
hipMemcpy(&COUNT615ba433d0a0, d_COUNT615ba433d0a0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT615ba433d0a0);
// Insert hash table control;
uint64_t* d_BUF_IDX_615ba433d0a0;
hipMalloc(&d_BUF_IDX_615ba433d0a0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_615ba433d0a0, 0, sizeof(uint64_t));
uint64_t* d_BUF_615ba433d0a0;
hipMalloc(&d_BUF_615ba433d0a0, sizeof(uint64_t) * COUNT615ba433d0a0 * 1);
auto d_HT_615ba433d0a0 = cuco::experimental::static_multimap{ (int)COUNT615ba433d0a0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_615ba423c2c0<<<std::ceil((float)part_size/32.), 32>>>(d_BUF_615ba433d0a0, d_BUF_IDX_615ba433d0a0, d_HT_615ba433d0a0.ref(cuco::insert), d_part__p_partkey, part_size);
hipFree(d_BUF_IDX_615ba433d0a0);
//Materialize count
uint64_t* d_COUNT615ba433d1b0;
hipMalloc(&d_COUNT615ba433d1b0, sizeof(uint64_t));
hipMemset(d_COUNT615ba433d1b0, 0, sizeof(uint64_t));
count_615ba43429c0<<<std::ceil((float)partsupp_size/32.), 32>>>(d_BUF_615ba4334780, d_COUNT615ba433d1b0, d_HT_615ba4334780.ref(cuco::for_each), d_partsupp__ps_suppkey, partsupp_size);
uint64_t COUNT615ba433d1b0;
hipMemcpy(&COUNT615ba433d1b0, d_COUNT615ba433d1b0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT615ba433d1b0);
// Insert hash table control;
uint64_t* d_BUF_IDX_615ba433d1b0;
hipMalloc(&d_BUF_IDX_615ba433d1b0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_615ba433d1b0, 0, sizeof(uint64_t));
uint64_t* d_BUF_615ba433d1b0;
hipMalloc(&d_BUF_615ba433d1b0, sizeof(uint64_t) * COUNT615ba433d1b0 * 3);
auto d_HT_615ba433d1b0 = cuco::experimental::static_multimap{ (int)COUNT615ba433d1b0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_615ba43429c0<<<std::ceil((float)partsupp_size/32.), 32>>>(d_BUF_615ba4334780, d_BUF_615ba433d1b0, d_BUF_IDX_615ba433d1b0, d_HT_615ba4334780.ref(cuco::for_each), d_HT_615ba433d1b0.ref(cuco::insert), d_partsupp__ps_partkey, d_partsupp__ps_suppkey, partsupp_size, d_supplier__s_suppkey);
hipFree(d_BUF_IDX_615ba433d1b0);
//Create aggregation hash table
auto d_HT_615ba42f1d30 = cuco::static_map{ (int)48009721*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_615ba4340570<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_615ba433cfe0, d_BUF_615ba433d0a0, d_BUF_615ba433d1b0, d_HT_615ba42f1d30.ref(cuco::insert), d_HT_615ba433cfe0.ref(cuco::for_each), d_HT_615ba433d0a0.ref(cuco::for_each), d_HT_615ba433d1b0.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_suppkey, lineitem_size, d_orders__o_orderdate);
size_t COUNT615ba42f1d30 = d_HT_615ba42f1d30.size();
thrust::device_vector<int64_t> keys_615ba42f1d30(COUNT615ba42f1d30), vals_615ba42f1d30(COUNT615ba42f1d30);
d_HT_615ba42f1d30.retrieve_all(keys_615ba42f1d30.begin(), vals_615ba42f1d30.begin());
d_HT_615ba42f1d30.clear();
int64_t* raw_keys615ba42f1d30 = thrust::raw_pointer_cast(keys_615ba42f1d30.data());
insertKeys<<<std::ceil((float)COUNT615ba42f1d30/32.), 32>>>(raw_keys615ba42f1d30, d_HT_615ba42f1d30.ref(cuco::insert), COUNT615ba42f1d30);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT615ba42f1d30);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT615ba42f1d30);
DBI64Type* d_KEY_615ba42f1d30map0__tmp_attr0;
hipMalloc(&d_KEY_615ba42f1d30map0__tmp_attr0, sizeof(DBI64Type) * COUNT615ba42f1d30);
hipMemset(d_KEY_615ba42f1d30map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT615ba42f1d30);
main_615ba4340570<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_615ba433cfe0, d_BUF_615ba433d0a0, d_BUF_615ba433d1b0, d_HT_615ba42f1d30.ref(cuco::find), d_HT_615ba433cfe0.ref(cuco::for_each), d_HT_615ba433d0a0.ref(cuco::for_each), d_HT_615ba433d1b0.ref(cuco::for_each), d_KEY_615ba42f1d30map0__tmp_attr0, d_aggr0__tmp_attr2, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_quantity, d_lineitem__l_suppkey, lineitem_size, d_orders__o_orderdate, d_partsupp__ps_supplycost);
//Materialize count
uint64_t* d_COUNT615ba43041f0;
hipMalloc(&d_COUNT615ba43041f0, sizeof(uint64_t));
hipMemset(d_COUNT615ba43041f0, 0, sizeof(uint64_t));
count_615ba4356b90<<<std::ceil((float)COUNT615ba42f1d30/32.), 32>>>(COUNT615ba42f1d30, d_COUNT615ba43041f0);
uint64_t COUNT615ba43041f0;
hipMemcpy(&COUNT615ba43041f0, d_COUNT615ba43041f0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT615ba43041f0);
//Materialize buffers
uint64_t* d_MAT_IDX615ba43041f0;
hipMalloc(&d_MAT_IDX615ba43041f0, sizeof(uint64_t));
hipMemset(d_MAT_IDX615ba43041f0, 0, sizeof(uint64_t));
auto MAT615ba43041f0map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT615ba43041f0);
DBI64Type* d_MAT615ba43041f0map0__tmp_attr0;
hipMalloc(&d_MAT615ba43041f0map0__tmp_attr0, sizeof(DBI64Type) * COUNT615ba43041f0);
auto MAT615ba43041f0aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT615ba43041f0);
DBDecimalType* d_MAT615ba43041f0aggr0__tmp_attr2;
hipMalloc(&d_MAT615ba43041f0aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT615ba43041f0);
main_615ba4356b90<<<std::ceil((float)COUNT615ba42f1d30/32.), 32>>>(COUNT615ba42f1d30, d_MAT615ba43041f0aggr0__tmp_attr2, d_MAT615ba43041f0map0__tmp_attr0, d_MAT_IDX615ba43041f0, d_aggr0__tmp_attr2, d_KEY_615ba42f1d30map0__tmp_attr0);
hipFree(d_MAT_IDX615ba43041f0);
hipMemcpy(MAT615ba43041f0map0__tmp_attr0, d_MAT615ba43041f0map0__tmp_attr0, sizeof(DBI64Type) * COUNT615ba43041f0, hipMemcpyDeviceToHost);
hipMemcpy(MAT615ba43041f0aggr0__tmp_attr2, d_MAT615ba43041f0aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT615ba43041f0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT615ba43041f0; i++) { std::cout << MAT615ba43041f0map0__tmp_attr0[i] << "\t";
std::cout << MAT615ba43041f0aggr0__tmp_attr2[i] << "\t";
std::cout << std::endl; }
}