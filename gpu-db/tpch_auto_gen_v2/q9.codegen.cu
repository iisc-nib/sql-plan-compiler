#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5b4bf5099a40(uint64_t* COUNT5b4bf508bf60, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
//Materialize count
atomicAdd((int*)COUNT5b4bf508bf60, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5b4bf5099a40(uint64_t* BUF_5b4bf508bf60, uint64_t* BUF_IDX_5b4bf508bf60, HASHTABLE_INSERT HT_5b4bf508bf60, DBI32Type* nation__n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_5b4bf508bf60 = 0;
auto reg_nation__n_nationkey = nation__n_nationkey[tid];

KEY_5b4bf508bf60 |= reg_nation__n_nationkey;
// Insert hash table kernel;
auto buf_idx_5b4bf508bf60 = atomicAdd((int*)BUF_IDX_5b4bf508bf60, 1);
HT_5b4bf508bf60.insert(cuco::pair{KEY_5b4bf508bf60, buf_idx_5b4bf508bf60});
BUF_5b4bf508bf60[buf_idx_5b4bf508bf60 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5b4bf5079010(uint64_t* BUF_5b4bf508bf60, uint64_t* COUNT5b4bf508c340, HASHTABLE_PROBE HT_5b4bf508bf60, DBI32Type* supplier__s_nationkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_5b4bf508bf60 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_5b4bf508bf60 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_5b4bf508bf60.for_each(KEY_5b4bf508bf60, [&] __device__ (auto const SLOT_5b4bf508bf60) {

auto const [slot_first5b4bf508bf60, slot_second5b4bf508bf60] = SLOT_5b4bf508bf60;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5b4bf508c340, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5b4bf5079010(uint64_t* BUF_5b4bf508bf60, uint64_t* BUF_5b4bf508c340, uint64_t* BUF_IDX_5b4bf508c340, HASHTABLE_PROBE HT_5b4bf508bf60, HASHTABLE_INSERT HT_5b4bf508c340, DBI32Type* supplier__s_nationkey, DBI32Type* supplier__s_suppkey, size_t supplier_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= supplier_size) return;
uint64_t KEY_5b4bf508bf60 = 0;
auto reg_supplier__s_nationkey = supplier__s_nationkey[tid];

KEY_5b4bf508bf60 |= reg_supplier__s_nationkey;
//Probe Hash table
HT_5b4bf508bf60.for_each(KEY_5b4bf508bf60, [&] __device__ (auto const SLOT_5b4bf508bf60) {
auto const [slot_first5b4bf508bf60, slot_second5b4bf508bf60] = SLOT_5b4bf508bf60;
if (!(true)) return;
uint64_t KEY_5b4bf508c340 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[tid];

KEY_5b4bf508c340 |= reg_supplier__s_suppkey;
// Insert hash table kernel;
auto buf_idx_5b4bf508c340 = atomicAdd((int*)BUF_IDX_5b4bf508c340, 1);
HT_5b4bf508c340.insert(cuco::pair{KEY_5b4bf508c340, buf_idx_5b4bf508c340});
BUF_5b4bf508c340[buf_idx_5b4bf508c340 * 2 + 0] = tid;
BUF_5b4bf508c340[buf_idx_5b4bf508c340 * 2 + 1] = BUF_5b4bf508bf60[slot_second5b4bf508bf60 * 1 + 0];
});
}
__global__ void count_5b4bf509f630(uint64_t* COUNT5b4bf508f980, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
//Materialize count
atomicAdd((int*)COUNT5b4bf508f980, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5b4bf509f630(uint64_t* BUF_5b4bf508f980, uint64_t* BUF_IDX_5b4bf508f980, HASHTABLE_INSERT HT_5b4bf508f980, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_5b4bf508f980 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_5b4bf508f980 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_5b4bf508f980 = atomicAdd((int*)BUF_IDX_5b4bf508f980, 1);
HT_5b4bf508f980.insert(cuco::pair{KEY_5b4bf508f980, buf_idx_5b4bf508f980});
BUF_5b4bf508f980[buf_idx_5b4bf508f980 * 1 + 0] = tid;
}
__global__ void count_5b4bf50789e0(uint64_t* COUNT5b4bf508fa90, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
//Materialize count
atomicAdd((int*)COUNT5b4bf508fa90, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5b4bf50789e0(uint64_t* BUF_5b4bf508fa90, uint64_t* BUF_IDX_5b4bf508fa90, HASHTABLE_INSERT HT_5b4bf508fa90, DBI32Type* part__p_partkey, size_t part_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= part_size) return;
uint64_t KEY_5b4bf508fa90 = 0;
auto reg_part__p_partkey = part__p_partkey[tid];

KEY_5b4bf508fa90 |= reg_part__p_partkey;
// Insert hash table kernel;
auto buf_idx_5b4bf508fa90 = atomicAdd((int*)BUF_IDX_5b4bf508fa90, 1);
HT_5b4bf508fa90.insert(cuco::pair{KEY_5b4bf508fa90, buf_idx_5b4bf508fa90});
BUF_5b4bf508fa90[buf_idx_5b4bf508fa90 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5b4bf5098b00(uint64_t* BUF_5b4bf508c340, uint64_t* COUNT5b4bf508fba0, HASHTABLE_PROBE HT_5b4bf508c340, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= partsupp_size) return;
uint64_t KEY_5b4bf508c340 = 0;
auto reg_partsupp__ps_suppkey = partsupp__ps_suppkey[tid];

KEY_5b4bf508c340 |= reg_partsupp__ps_suppkey;
//Probe Hash table
HT_5b4bf508c340.for_each(KEY_5b4bf508c340, [&] __device__ (auto const SLOT_5b4bf508c340) {

auto const [slot_first5b4bf508c340, slot_second5b4bf508c340] = SLOT_5b4bf508c340;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5b4bf508fba0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5b4bf5098b00(uint64_t* BUF_5b4bf508c340, uint64_t* BUF_5b4bf508fba0, uint64_t* BUF_IDX_5b4bf508fba0, HASHTABLE_PROBE HT_5b4bf508c340, HASHTABLE_INSERT HT_5b4bf508fba0, DBI32Type* partsupp__ps_partkey, DBI32Type* partsupp__ps_suppkey, size_t partsupp_size, DBI32Type* supplier__s_suppkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= partsupp_size) return;
uint64_t KEY_5b4bf508c340 = 0;
auto reg_partsupp__ps_suppkey = partsupp__ps_suppkey[tid];

KEY_5b4bf508c340 |= reg_partsupp__ps_suppkey;
//Probe Hash table
HT_5b4bf508c340.for_each(KEY_5b4bf508c340, [&] __device__ (auto const SLOT_5b4bf508c340) {
auto const [slot_first5b4bf508c340, slot_second5b4bf508c340] = SLOT_5b4bf508c340;
if (!(true)) return;
uint64_t KEY_5b4bf508fba0 = 0;
auto reg_supplier__s_suppkey = supplier__s_suppkey[BUF_5b4bf508c340[slot_second5b4bf508c340 * 2 + 0]];

KEY_5b4bf508fba0 |= reg_supplier__s_suppkey;
auto reg_partsupp__ps_partkey = partsupp__ps_partkey[tid];
KEY_5b4bf508fba0 <<= 32;
KEY_5b4bf508fba0 |= reg_partsupp__ps_partkey;
// Insert hash table kernel;
auto buf_idx_5b4bf508fba0 = atomicAdd((int*)BUF_IDX_5b4bf508fba0, 1);
HT_5b4bf508fba0.insert(cuco::pair{KEY_5b4bf508fba0, buf_idx_5b4bf508fba0});
BUF_5b4bf508fba0[buf_idx_5b4bf508fba0 * 3 + 0] = BUF_5b4bf508c340[slot_second5b4bf508c340 * 2 + 0];
BUF_5b4bf508fba0[buf_idx_5b4bf508fba0 * 3 + 1] = tid;
BUF_5b4bf508fba0[buf_idx_5b4bf508fba0 * 3 + 2] = BUF_5b4bf508c340[slot_second5b4bf508c340 * 2 + 1];
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5b4bf5096650(uint64_t* BUF_5b4bf508f980, uint64_t* BUF_5b4bf508fa90, uint64_t* BUF_5b4bf508fba0, HASHTABLE_INSERT HT_5b4bf5048910, HASHTABLE_PROBE HT_5b4bf508f980, HASHTABLE_PROBE HT_5b4bf508fa90, HASHTABLE_PROBE HT_5b4bf508fba0, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBDateType* orders__o_orderdate) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_5b4bf508f980 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5b4bf508f980 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5b4bf508f980.for_each(KEY_5b4bf508f980, [&] __device__ (auto const SLOT_5b4bf508f980) {

auto const [slot_first5b4bf508f980, slot_second5b4bf508f980] = SLOT_5b4bf508f980;
if (!(true)) return;
uint64_t KEY_5b4bf508fa90 = 0;
auto reg_lineitem__l_partkey = lineitem__l_partkey[tid];

KEY_5b4bf508fa90 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_5b4bf508fa90.for_each(KEY_5b4bf508fa90, [&] __device__ (auto const SLOT_5b4bf508fa90) {

auto const [slot_first5b4bf508fa90, slot_second5b4bf508fa90] = SLOT_5b4bf508fa90;
if (!(true)) return;
uint64_t KEY_5b4bf508fba0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_5b4bf508fba0 |= reg_lineitem__l_suppkey;
KEY_5b4bf508fba0 <<= 32;
KEY_5b4bf508fba0 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_5b4bf508fba0.for_each(KEY_5b4bf508fba0, [&] __device__ (auto const SLOT_5b4bf508fba0) {

auto const [slot_first5b4bf508fba0, slot_second5b4bf508fba0] = SLOT_5b4bf508fba0;
if (!(true)) return;
uint64_t KEY_5b4bf5048910 = 0;
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_5b4bf508f980[slot_second5b4bf508f980 * 1 + 0]];
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_orders__o_orderdate);

KEY_5b4bf5048910 |= reg_map0__tmp_attr0;
//Create aggregation hash table
HT_5b4bf5048910.insert(cuco::pair{KEY_5b4bf5048910, 1});
});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5b4bf5096650(uint64_t* BUF_5b4bf508f980, uint64_t* BUF_5b4bf508fa90, uint64_t* BUF_5b4bf508fba0, HASHTABLE_FIND HT_5b4bf5048910, HASHTABLE_PROBE HT_5b4bf508f980, HASHTABLE_PROBE HT_5b4bf508fa90, HASHTABLE_PROBE HT_5b4bf508fba0, DBI64Type* KEY_5b4bf5048910map0__tmp_attr0, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBI32Type* lineitem__l_partkey, DBDecimalType* lineitem__l_quantity, DBI32Type* lineitem__l_suppkey, size_t lineitem_size, DBDateType* orders__o_orderdate, DBDecimalType* partsupp__ps_supplycost) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
uint64_t KEY_5b4bf508f980 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5b4bf508f980 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5b4bf508f980.for_each(KEY_5b4bf508f980, [&] __device__ (auto const SLOT_5b4bf508f980) {
auto const [slot_first5b4bf508f980, slot_second5b4bf508f980] = SLOT_5b4bf508f980;
if (!(true)) return;
uint64_t KEY_5b4bf508fa90 = 0;
auto reg_lineitem__l_partkey = lineitem__l_partkey[tid];

KEY_5b4bf508fa90 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_5b4bf508fa90.for_each(KEY_5b4bf508fa90, [&] __device__ (auto const SLOT_5b4bf508fa90) {
auto const [slot_first5b4bf508fa90, slot_second5b4bf508fa90] = SLOT_5b4bf508fa90;
if (!(true)) return;
uint64_t KEY_5b4bf508fba0 = 0;
auto reg_lineitem__l_suppkey = lineitem__l_suppkey[tid];

KEY_5b4bf508fba0 |= reg_lineitem__l_suppkey;
KEY_5b4bf508fba0 <<= 32;
KEY_5b4bf508fba0 |= reg_lineitem__l_partkey;
//Probe Hash table
HT_5b4bf508fba0.for_each(KEY_5b4bf508fba0, [&] __device__ (auto const SLOT_5b4bf508fba0) {
auto const [slot_first5b4bf508fba0, slot_second5b4bf508fba0] = SLOT_5b4bf508fba0;
if (!(true)) return;
uint64_t KEY_5b4bf5048910 = 0;
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_5b4bf508f980[slot_second5b4bf508f980 * 1 + 0]];
auto reg_map0__tmp_attr0 = ExtractFromDate("year", reg_orders__o_orderdate);

KEY_5b4bf5048910 |= reg_map0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_5b4bf5048910 = HT_5b4bf5048910.find(KEY_5b4bf5048910)->second;
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
auto reg_partsupp__ps_supplycost = partsupp__ps_supplycost[BUF_5b4bf508fba0[slot_second5b4bf508fba0 * 3 + 1]];
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = ((reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount))) - ((reg_partsupp__ps_supplycost) * (reg_lineitem__l_quantity));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_5b4bf5048910], reg_map0__tmp_attr1);
KEY_5b4bf5048910map0__tmp_attr0[buf_idx_5b4bf5048910] = reg_map0__tmp_attr0;
});
});
});
}
__global__ void count_5b4bf50ac720(size_t COUNT5b4bf5048910, uint64_t* COUNT5b4bf505bb00) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5b4bf5048910) return;
//Materialize count
atomicAdd((int*)COUNT5b4bf505bb00, 1);
}
__global__ void main_5b4bf50ac720(size_t COUNT5b4bf5048910, DBDecimalType* MAT5b4bf505bb00aggr0__tmp_attr2, DBI64Type* MAT5b4bf505bb00map0__tmp_attr0, uint64_t* MAT_IDX5b4bf505bb00, DBDecimalType* aggr0__tmp_attr2, DBI64Type* map0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5b4bf5048910) return;
//Materialize buffers
auto mat_idx5b4bf505bb00 = atomicAdd((int*)MAT_IDX5b4bf505bb00, 1);
auto reg_map0__tmp_attr0 = map0__tmp_attr0[tid];
MAT5b4bf505bb00map0__tmp_attr0[mat_idx5b4bf505bb00] = reg_map0__tmp_attr0;
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT5b4bf505bb00aggr0__tmp_attr2[mat_idx5b4bf505bb00] = reg_aggr0__tmp_attr2;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT5b4bf508bf60;
hipMalloc(&d_COUNT5b4bf508bf60, sizeof(uint64_t));
hipMemset(d_COUNT5b4bf508bf60, 0, sizeof(uint64_t));
count_5b4bf5099a40<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT5b4bf508bf60, nation_size);
uint64_t COUNT5b4bf508bf60;
hipMemcpy(&COUNT5b4bf508bf60, d_COUNT5b4bf508bf60, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b4bf508bf60;
hipMalloc(&d_BUF_IDX_5b4bf508bf60, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b4bf508bf60, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b4bf508bf60;
hipMalloc(&d_BUF_5b4bf508bf60, sizeof(uint64_t) * COUNT5b4bf508bf60 * 1);
auto d_HT_5b4bf508bf60 = cuco::experimental::static_multimap{ (int)COUNT5b4bf508bf60*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b4bf5099a40<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_5b4bf508bf60, d_BUF_IDX_5b4bf508bf60, d_HT_5b4bf508bf60.ref(cuco::insert), d_nation__n_nationkey, nation_size);
//Materialize count
uint64_t* d_COUNT5b4bf508c340;
hipMalloc(&d_COUNT5b4bf508c340, sizeof(uint64_t));
hipMemset(d_COUNT5b4bf508c340, 0, sizeof(uint64_t));
count_5b4bf5079010<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_5b4bf508bf60, d_COUNT5b4bf508c340, d_HT_5b4bf508bf60.ref(cuco::for_each), d_supplier__s_nationkey, supplier_size);
uint64_t COUNT5b4bf508c340;
hipMemcpy(&COUNT5b4bf508c340, d_COUNT5b4bf508c340, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b4bf508c340;
hipMalloc(&d_BUF_IDX_5b4bf508c340, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b4bf508c340, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b4bf508c340;
hipMalloc(&d_BUF_5b4bf508c340, sizeof(uint64_t) * COUNT5b4bf508c340 * 2);
auto d_HT_5b4bf508c340 = cuco::experimental::static_multimap{ (int)COUNT5b4bf508c340*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b4bf5079010<<<std::ceil((float)supplier_size/32.), 32>>>(d_BUF_5b4bf508bf60, d_BUF_5b4bf508c340, d_BUF_IDX_5b4bf508c340, d_HT_5b4bf508bf60.ref(cuco::for_each), d_HT_5b4bf508c340.ref(cuco::insert), d_supplier__s_nationkey, d_supplier__s_suppkey, supplier_size);
//Materialize count
uint64_t* d_COUNT5b4bf508f980;
hipMalloc(&d_COUNT5b4bf508f980, sizeof(uint64_t));
hipMemset(d_COUNT5b4bf508f980, 0, sizeof(uint64_t));
count_5b4bf509f630<<<std::ceil((float)orders_size/32.), 32>>>(d_COUNT5b4bf508f980, orders_size);
uint64_t COUNT5b4bf508f980;
hipMemcpy(&COUNT5b4bf508f980, d_COUNT5b4bf508f980, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b4bf508f980;
hipMalloc(&d_BUF_IDX_5b4bf508f980, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b4bf508f980, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b4bf508f980;
hipMalloc(&d_BUF_5b4bf508f980, sizeof(uint64_t) * COUNT5b4bf508f980 * 1);
auto d_HT_5b4bf508f980 = cuco::experimental::static_multimap{ (int)COUNT5b4bf508f980*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b4bf509f630<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5b4bf508f980, d_BUF_IDX_5b4bf508f980, d_HT_5b4bf508f980.ref(cuco::insert), d_orders__o_orderkey, orders_size);
//Materialize count
uint64_t* d_COUNT5b4bf508fa90;
hipMalloc(&d_COUNT5b4bf508fa90, sizeof(uint64_t));
hipMemset(d_COUNT5b4bf508fa90, 0, sizeof(uint64_t));
count_5b4bf50789e0<<<std::ceil((float)part_size/32.), 32>>>(d_COUNT5b4bf508fa90, part_size);
uint64_t COUNT5b4bf508fa90;
hipMemcpy(&COUNT5b4bf508fa90, d_COUNT5b4bf508fa90, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b4bf508fa90;
hipMalloc(&d_BUF_IDX_5b4bf508fa90, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b4bf508fa90, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b4bf508fa90;
hipMalloc(&d_BUF_5b4bf508fa90, sizeof(uint64_t) * COUNT5b4bf508fa90 * 1);
auto d_HT_5b4bf508fa90 = cuco::experimental::static_multimap{ (int)COUNT5b4bf508fa90*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b4bf50789e0<<<std::ceil((float)part_size/32.), 32>>>(d_BUF_5b4bf508fa90, d_BUF_IDX_5b4bf508fa90, d_HT_5b4bf508fa90.ref(cuco::insert), d_part__p_partkey, part_size);
//Materialize count
uint64_t* d_COUNT5b4bf508fba0;
hipMalloc(&d_COUNT5b4bf508fba0, sizeof(uint64_t));
hipMemset(d_COUNT5b4bf508fba0, 0, sizeof(uint64_t));
count_5b4bf5098b00<<<std::ceil((float)partsupp_size/32.), 32>>>(d_BUF_5b4bf508c340, d_COUNT5b4bf508fba0, d_HT_5b4bf508c340.ref(cuco::for_each), d_partsupp__ps_suppkey, partsupp_size);
uint64_t COUNT5b4bf508fba0;
hipMemcpy(&COUNT5b4bf508fba0, d_COUNT5b4bf508fba0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b4bf508fba0;
hipMalloc(&d_BUF_IDX_5b4bf508fba0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b4bf508fba0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b4bf508fba0;
hipMalloc(&d_BUF_5b4bf508fba0, sizeof(uint64_t) * COUNT5b4bf508fba0 * 3);
auto d_HT_5b4bf508fba0 = cuco::experimental::static_multimap{ (int)COUNT5b4bf508fba0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b4bf5098b00<<<std::ceil((float)partsupp_size/32.), 32>>>(d_BUF_5b4bf508c340, d_BUF_5b4bf508fba0, d_BUF_IDX_5b4bf508fba0, d_HT_5b4bf508c340.ref(cuco::for_each), d_HT_5b4bf508fba0.ref(cuco::insert), d_partsupp__ps_partkey, d_partsupp__ps_suppkey, partsupp_size, d_supplier__s_suppkey);
//Create aggregation hash table
auto d_HT_5b4bf5048910 = cuco::static_map{ (int)48009721*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5b4bf5096650<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5b4bf508f980, d_BUF_5b4bf508fa90, d_BUF_5b4bf508fba0, d_HT_5b4bf5048910.ref(cuco::insert), d_HT_5b4bf508f980.ref(cuco::for_each), d_HT_5b4bf508fa90.ref(cuco::for_each), d_HT_5b4bf508fba0.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_suppkey, lineitem_size, d_orders__o_orderdate);
size_t COUNT5b4bf5048910 = d_HT_5b4bf5048910.size();
thrust::device_vector<int64_t> keys_5b4bf5048910(COUNT5b4bf5048910), vals_5b4bf5048910(COUNT5b4bf5048910);
d_HT_5b4bf5048910.retrieve_all(keys_5b4bf5048910.begin(), vals_5b4bf5048910.begin());
d_HT_5b4bf5048910.clear();
int64_t* raw_keys5b4bf5048910 = thrust::raw_pointer_cast(keys_5b4bf5048910.data());
insertKeys<<<std::ceil((float)COUNT5b4bf5048910/32.), 32>>>(raw_keys5b4bf5048910, d_HT_5b4bf5048910.ref(cuco::insert), COUNT5b4bf5048910);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT5b4bf5048910);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT5b4bf5048910);
DBI64Type* d_KEY_5b4bf5048910map0__tmp_attr0;
hipMalloc(&d_KEY_5b4bf5048910map0__tmp_attr0, sizeof(DBI64Type) * COUNT5b4bf5048910);
hipMemset(d_KEY_5b4bf5048910map0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT5b4bf5048910);
main_5b4bf5096650<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5b4bf508f980, d_BUF_5b4bf508fa90, d_BUF_5b4bf508fba0, d_HT_5b4bf5048910.ref(cuco::find), d_HT_5b4bf508f980.ref(cuco::for_each), d_HT_5b4bf508fa90.ref(cuco::for_each), d_HT_5b4bf508fba0.ref(cuco::for_each), d_KEY_5b4bf5048910map0__tmp_attr0, d_aggr0__tmp_attr2, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_partkey, d_lineitem__l_quantity, d_lineitem__l_suppkey, lineitem_size, d_orders__o_orderdate, d_partsupp__ps_supplycost);
//Materialize count
uint64_t* d_COUNT5b4bf505bb00;
hipMalloc(&d_COUNT5b4bf505bb00, sizeof(uint64_t));
hipMemset(d_COUNT5b4bf505bb00, 0, sizeof(uint64_t));
count_5b4bf50ac720<<<std::ceil((float)COUNT5b4bf5048910/32.), 32>>>(COUNT5b4bf5048910, d_COUNT5b4bf505bb00);
uint64_t COUNT5b4bf505bb00;
hipMemcpy(&COUNT5b4bf505bb00, d_COUNT5b4bf505bb00, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5b4bf505bb00;
hipMalloc(&d_MAT_IDX5b4bf505bb00, sizeof(uint64_t));
hipMemset(d_MAT_IDX5b4bf505bb00, 0, sizeof(uint64_t));
auto MAT5b4bf505bb00map0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT5b4bf505bb00);
DBI64Type* d_MAT5b4bf505bb00map0__tmp_attr0;
hipMalloc(&d_MAT5b4bf505bb00map0__tmp_attr0, sizeof(DBI64Type) * COUNT5b4bf505bb00);
auto MAT5b4bf505bb00aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5b4bf505bb00);
DBDecimalType* d_MAT5b4bf505bb00aggr0__tmp_attr2;
hipMalloc(&d_MAT5b4bf505bb00aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT5b4bf505bb00);
main_5b4bf50ac720<<<std::ceil((float)COUNT5b4bf5048910/32.), 32>>>(COUNT5b4bf5048910, d_MAT5b4bf505bb00aggr0__tmp_attr2, d_MAT5b4bf505bb00map0__tmp_attr0, d_MAT_IDX5b4bf505bb00, d_aggr0__tmp_attr2, d_KEY_5b4bf5048910map0__tmp_attr0);
hipMemcpy(MAT5b4bf505bb00map0__tmp_attr0, d_MAT5b4bf505bb00map0__tmp_attr0, sizeof(DBI64Type) * COUNT5b4bf505bb00, hipMemcpyDeviceToHost);
hipMemcpy(MAT5b4bf505bb00aggr0__tmp_attr2, d_MAT5b4bf505bb00aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT5b4bf505bb00, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5b4bf505bb00; i++) { std::cout << MAT5b4bf505bb00map0__tmp_attr0[i] << "\t";
std::cout << MAT5b4bf505bb00aggr0__tmp_attr2[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_5b4bf508bf60);
hipFree(d_BUF_IDX_5b4bf508bf60);
hipFree(d_COUNT5b4bf508bf60);
hipFree(d_BUF_5b4bf508c340);
hipFree(d_BUF_IDX_5b4bf508c340);
hipFree(d_COUNT5b4bf508c340);
hipFree(d_BUF_5b4bf508f980);
hipFree(d_BUF_IDX_5b4bf508f980);
hipFree(d_COUNT5b4bf508f980);
hipFree(d_BUF_5b4bf508fa90);
hipFree(d_BUF_IDX_5b4bf508fa90);
hipFree(d_COUNT5b4bf508fa90);
hipFree(d_BUF_5b4bf508fba0);
hipFree(d_BUF_IDX_5b4bf508fba0);
hipFree(d_COUNT5b4bf508fba0);
hipFree(d_KEY_5b4bf5048910map0__tmp_attr0);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_COUNT5b4bf505bb00);
hipFree(d_MAT5b4bf505bb00aggr0__tmp_attr2);
hipFree(d_MAT5b4bf505bb00map0__tmp_attr0);
hipFree(d_MAT_IDX5b4bf505bb00);
free(MAT5b4bf505bb00aggr0__tmp_attr2);
free(MAT5b4bf505bb00map0__tmp_attr0);
}