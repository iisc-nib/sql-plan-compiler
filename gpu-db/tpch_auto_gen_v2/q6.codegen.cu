#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
template<typename HASHTABLE_INSERT>
__global__ void count_6385a4f00800(HASHTABLE_INSERT HT_6385a4eb7ee0, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_quantity, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::lt))) return;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
if (!(evaluatePredicate(reg_lineitem__l_discount, 0.05, Predicate::gte) && evaluatePredicate(reg_lineitem__l_discount, 0.07, Predicate::lte))) return;
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
if (!(evaluatePredicate(reg_lineitem__l_quantity, 24.0, Predicate::lt))) return;
uint64_t KEY_6385a4eb7ee0 = 0;
//Create aggregation hash table
HT_6385a4eb7ee0.insert(cuco::pair{KEY_6385a4eb7ee0, 1});
}
template<typename HASHTABLE_FIND>
__global__ void main_6385a4f00800(HASHTABLE_FIND HT_6385a4eb7ee0, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBDecimalType* lineitem__l_quantity, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::lt))) return;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
if (!(evaluatePredicate(reg_lineitem__l_discount, 0.05, Predicate::gte) && evaluatePredicate(reg_lineitem__l_discount, 0.07, Predicate::lte))) return;
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
if (!(evaluatePredicate(reg_lineitem__l_quantity, 24.0, Predicate::lt))) return;
uint64_t KEY_6385a4eb7ee0 = 0;
//Aggregate in hashtable
auto buf_idx_6385a4eb7ee0 = HT_6385a4eb7ee0.find(KEY_6385a4eb7ee0)->second;
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * (reg_lineitem__l_discount);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_6385a4eb7ee0], reg_map0__tmp_attr1);
}
__global__ void count_6385a4f01ee0(size_t COUNT6385a4eb7ee0, uint64_t* COUNT6385a4ecb4b0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT6385a4eb7ee0) return;
//Materialize count
atomicAdd((int*)COUNT6385a4ecb4b0, 1);
}
__global__ void main_6385a4f01ee0(size_t COUNT6385a4eb7ee0, DBDecimalType* MAT6385a4ecb4b0aggr0__tmp_attr0, uint64_t* MAT_IDX6385a4ecb4b0, DBDecimalType* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT6385a4eb7ee0) return;
//Materialize buffers
auto mat_idx6385a4ecb4b0 = atomicAdd((int*)MAT_IDX6385a4ecb4b0, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT6385a4ecb4b0aggr0__tmp_attr0[mat_idx6385a4ecb4b0] = reg_aggr0__tmp_attr0;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Create aggregation hash table
auto d_HT_6385a4eb7ee0 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_6385a4f00800<<<std::ceil((float)lineitem_size/32.), 32>>>(d_HT_6385a4eb7ee0.ref(cuco::insert), d_lineitem__l_discount, d_lineitem__l_quantity, d_lineitem__l_shipdate, lineitem_size);
size_t COUNT6385a4eb7ee0 = d_HT_6385a4eb7ee0.size();
thrust::device_vector<int64_t> keys_6385a4eb7ee0(COUNT6385a4eb7ee0), vals_6385a4eb7ee0(COUNT6385a4eb7ee0);
d_HT_6385a4eb7ee0.retrieve_all(keys_6385a4eb7ee0.begin(), vals_6385a4eb7ee0.begin());
d_HT_6385a4eb7ee0.clear();
int64_t* raw_keys6385a4eb7ee0 = thrust::raw_pointer_cast(keys_6385a4eb7ee0.data());
insertKeys<<<std::ceil((float)COUNT6385a4eb7ee0/32.), 32>>>(raw_keys6385a4eb7ee0, d_HT_6385a4eb7ee0.ref(cuco::insert), COUNT6385a4eb7ee0);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6385a4eb7ee0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT6385a4eb7ee0);
main_6385a4f00800<<<std::ceil((float)lineitem_size/32.), 32>>>(d_HT_6385a4eb7ee0.ref(cuco::find), d_aggr0__tmp_attr0, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_quantity, d_lineitem__l_shipdate, lineitem_size);
//Materialize count
uint64_t* d_COUNT6385a4ecb4b0;
hipMalloc(&d_COUNT6385a4ecb4b0, sizeof(uint64_t));
hipMemset(d_COUNT6385a4ecb4b0, 0, sizeof(uint64_t));
count_6385a4f01ee0<<<std::ceil((float)COUNT6385a4eb7ee0/32.), 32>>>(COUNT6385a4eb7ee0, d_COUNT6385a4ecb4b0);
uint64_t COUNT6385a4ecb4b0;
hipMemcpy(&COUNT6385a4ecb4b0, d_COUNT6385a4ecb4b0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX6385a4ecb4b0;
hipMalloc(&d_MAT_IDX6385a4ecb4b0, sizeof(uint64_t));
hipMemset(d_MAT_IDX6385a4ecb4b0, 0, sizeof(uint64_t));
auto MAT6385a4ecb4b0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT6385a4ecb4b0);
DBDecimalType* d_MAT6385a4ecb4b0aggr0__tmp_attr0;
hipMalloc(&d_MAT6385a4ecb4b0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6385a4ecb4b0);
main_6385a4f01ee0<<<std::ceil((float)COUNT6385a4eb7ee0/32.), 32>>>(COUNT6385a4eb7ee0, d_MAT6385a4ecb4b0aggr0__tmp_attr0, d_MAT_IDX6385a4ecb4b0, d_aggr0__tmp_attr0);
hipMemcpy(MAT6385a4ecb4b0aggr0__tmp_attr0, d_MAT6385a4ecb4b0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT6385a4ecb4b0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT6385a4ecb4b0; i++) { std::cout << MAT6385a4ecb4b0aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
hipFree(d_aggr0__tmp_attr0);
hipFree(d_COUNT6385a4ecb4b0);
hipFree(d_MAT6385a4ecb4b0aggr0__tmp_attr0);
hipFree(d_MAT_IDX6385a4ecb4b0);
free(MAT6385a4ecb4b0aggr0__tmp_attr0);
}