#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
template<typename HASHTABLE_INSERT>
__global__ void count_62f8e8faa090(HASHTABLE_INSERT HT_62f8e9056380, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_quantity, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::lt))) return;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
if (!(evaluatePredicate(reg_lineitem__l_discount, 0.05, Predicate::gte) && evaluatePredicate(reg_lineitem__l_discount, 0.07, Predicate::lte))) return;
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
if (!(evaluatePredicate(reg_lineitem__l_quantity, 24., Predicate::lt))) return;
uint64_t KEY_62f8e9056380 = 0;
//Create aggregation hash table
HT_62f8e9056380.insert(cuco::pair{KEY_62f8e9056380, 1});
}
template<typename HASHTABLE_FIND>
__global__ void main_62f8e8faa090(HASHTABLE_FIND HT_62f8e9056380, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBDecimalType* lineitem__l_quantity, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::lt))) return;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
if (!(evaluatePredicate(reg_lineitem__l_discount, 0.05, Predicate::gte) && evaluatePredicate(reg_lineitem__l_discount, 0.07, Predicate::lte))) return;
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
if (!(evaluatePredicate(reg_lineitem__l_quantity, 24., Predicate::lt))) return;
uint64_t KEY_62f8e9056380 = 0;
//Aggregate in hashtable
auto buf_idx_62f8e9056380 = HT_62f8e9056380.find(KEY_62f8e9056380)->second;
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * (reg_lineitem__l_discount);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_62f8e9056380], reg_map0__tmp_attr1);
}
__global__ void count_62f8e902c890(size_t COUNT62f8e9056380, uint64_t* COUNT62f8e90687e0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT62f8e9056380) return;
//Materialize count
atomicAdd((int*)COUNT62f8e90687e0, 1);
}
__global__ void main_62f8e902c890(size_t COUNT62f8e9056380, DBDecimalType* MAT62f8e90687e0aggr0__tmp_attr0, uint64_t* MAT_IDX62f8e90687e0, DBDecimalType* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT62f8e9056380) return;
//Materialize buffers
auto mat_idx62f8e90687e0 = atomicAdd((int*)MAT_IDX62f8e90687e0, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT62f8e90687e0aggr0__tmp_attr0[mat_idx62f8e90687e0] = reg_aggr0__tmp_attr0;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Create aggregation hash table
auto d_HT_62f8e9056380 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_62f8e8faa090<<<std::ceil((float)lineitem_size/32.), 32>>>(d_HT_62f8e9056380.ref(cuco::insert), d_lineitem__l_discount, d_lineitem__l_quantity, d_lineitem__l_shipdate, lineitem_size);
size_t COUNT62f8e9056380 = d_HT_62f8e9056380.size();
thrust::device_vector<int64_t> keys_62f8e9056380(COUNT62f8e9056380), vals_62f8e9056380(COUNT62f8e9056380);
d_HT_62f8e9056380.retrieve_all(keys_62f8e9056380.begin(), vals_62f8e9056380.begin());
thrust::host_vector<int64_t> h_keys_62f8e9056380(COUNT62f8e9056380);
thrust::copy(keys_62f8e9056380.begin(), keys_62f8e9056380.end(), h_keys_62f8e9056380.begin());
thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_62f8e9056380(COUNT62f8e9056380);
for (int i=0; i < COUNT62f8e9056380; i++)
{actual_dict_62f8e9056380[i] = cuco::make_pair(h_keys_62f8e9056380[i], i);}
d_HT_62f8e9056380.clear();
d_HT_62f8e9056380.insert(actual_dict_62f8e9056380.begin(), actual_dict_62f8e9056380.end());
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT62f8e9056380);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT62f8e9056380);
main_62f8e8faa090<<<std::ceil((float)lineitem_size/32.), 32>>>(d_HT_62f8e9056380.ref(cuco::find), d_aggr0__tmp_attr0, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_quantity, d_lineitem__l_shipdate, lineitem_size);
//Materialize count
uint64_t* d_COUNT62f8e90687e0;
hipMalloc(&d_COUNT62f8e90687e0, sizeof(uint64_t));
hipMemset(d_COUNT62f8e90687e0, 0, sizeof(uint64_t));
count_62f8e902c890<<<std::ceil((float)COUNT62f8e9056380/32.), 32>>>(COUNT62f8e9056380, d_COUNT62f8e90687e0);
uint64_t COUNT62f8e90687e0;
hipMemcpy(&COUNT62f8e90687e0, d_COUNT62f8e90687e0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT62f8e90687e0);
//Materialize buffers
uint64_t* d_MAT_IDX62f8e90687e0;
hipMalloc(&d_MAT_IDX62f8e90687e0, sizeof(uint64_t));
hipMemset(d_MAT_IDX62f8e90687e0, 0, sizeof(uint64_t));
auto MAT62f8e90687e0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT62f8e90687e0);
DBDecimalType* d_MAT62f8e90687e0aggr0__tmp_attr0;
hipMalloc(&d_MAT62f8e90687e0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT62f8e90687e0);
main_62f8e902c890<<<std::ceil((float)COUNT62f8e9056380/32.), 32>>>(COUNT62f8e9056380, d_MAT62f8e90687e0aggr0__tmp_attr0, d_MAT_IDX62f8e90687e0, d_aggr0__tmp_attr0);
hipFree(d_MAT_IDX62f8e90687e0);
hipMemcpy(MAT62f8e90687e0aggr0__tmp_attr0, d_MAT62f8e90687e0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT62f8e90687e0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT62f8e90687e0; i++) { std::cout << MAT62f8e90687e0aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
}