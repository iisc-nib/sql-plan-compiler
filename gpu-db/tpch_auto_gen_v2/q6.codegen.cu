#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
template<typename HASHTABLE_INSERT>
__global__ void count_61e04de0fb10(HASHTABLE_INSERT HT_61e04debb940, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_quantity, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::lt))) return;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
if (!(evaluatePredicate(reg_lineitem__l_discount, 0.05, Predicate::gte) && evaluatePredicate(reg_lineitem__l_discount, 0.07, Predicate::lte))) return;
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
if (!(evaluatePredicate(reg_lineitem__l_quantity, 24.0, Predicate::lt))) return;
uint64_t KEY_61e04debb940 = 0;
//Create aggregation hash table
HT_61e04debb940.insert(cuco::pair{KEY_61e04debb940, 1});
}
template<typename HASHTABLE_FIND>
__global__ void main_61e04de0fb10(HASHTABLE_FIND HT_61e04debb940, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBDecimalType* lineitem__l_quantity, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 8766, Predicate::gte) && evaluatePredicate(reg_lineitem__l_shipdate, 9131, Predicate::lt))) return;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
if (!(evaluatePredicate(reg_lineitem__l_discount, 0.05, Predicate::gte) && evaluatePredicate(reg_lineitem__l_discount, 0.07, Predicate::lte))) return;
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
if (!(evaluatePredicate(reg_lineitem__l_quantity, 24.0, Predicate::lt))) return;
uint64_t KEY_61e04debb940 = 0;
//Aggregate in hashtable
auto buf_idx_61e04debb940 = HT_61e04debb940.find(KEY_61e04debb940)->second;
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * (reg_lineitem__l_discount);
aggregate_sum(&aggr0__tmp_attr0[buf_idx_61e04debb940], reg_map0__tmp_attr1);
}
__global__ void count_61e04de92310(size_t COUNT61e04debb940, uint64_t* COUNT61e04decd970) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT61e04debb940) return;
//Materialize count
atomicAdd((int*)COUNT61e04decd970, 1);
}
__global__ void main_61e04de92310(size_t COUNT61e04debb940, DBDecimalType* MAT61e04decd970aggr0__tmp_attr0, uint64_t* MAT_IDX61e04decd970, DBDecimalType* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT61e04debb940) return;
//Materialize buffers
auto mat_idx61e04decd970 = atomicAdd((int*)MAT_IDX61e04decd970, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT61e04decd970aggr0__tmp_attr0[mat_idx61e04decd970] = reg_aggr0__tmp_attr0;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Create aggregation hash table
auto d_HT_61e04debb940 = cuco::static_map{ (int)1*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_61e04de0fb10<<<std::ceil((float)lineitem_size/32.), 32>>>(d_HT_61e04debb940.ref(cuco::insert), d_lineitem__l_discount, d_lineitem__l_quantity, d_lineitem__l_shipdate, lineitem_size);
size_t COUNT61e04debb940 = d_HT_61e04debb940.size();
thrust::device_vector<int64_t> keys_61e04debb940(COUNT61e04debb940), vals_61e04debb940(COUNT61e04debb940);
d_HT_61e04debb940.retrieve_all(keys_61e04debb940.begin(), vals_61e04debb940.begin());
d_HT_61e04debb940.clear();
int64_t* raw_keys61e04debb940 = thrust::raw_pointer_cast(keys_61e04debb940.data());
insertKeys<<<std::ceil((float)COUNT61e04debb940/32.), 32>>>(raw_keys61e04debb940, d_HT_61e04debb940.ref(cuco::insert), COUNT61e04debb940);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT61e04debb940);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT61e04debb940);
main_61e04de0fb10<<<std::ceil((float)lineitem_size/32.), 32>>>(d_HT_61e04debb940.ref(cuco::find), d_aggr0__tmp_attr0, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_quantity, d_lineitem__l_shipdate, lineitem_size);
//Materialize count
uint64_t* d_COUNT61e04decd970;
hipMalloc(&d_COUNT61e04decd970, sizeof(uint64_t));
hipMemset(d_COUNT61e04decd970, 0, sizeof(uint64_t));
count_61e04de92310<<<std::ceil((float)COUNT61e04debb940/32.), 32>>>(COUNT61e04debb940, d_COUNT61e04decd970);
uint64_t COUNT61e04decd970;
hipMemcpy(&COUNT61e04decd970, d_COUNT61e04decd970, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT61e04decd970);
//Materialize buffers
uint64_t* d_MAT_IDX61e04decd970;
hipMalloc(&d_MAT_IDX61e04decd970, sizeof(uint64_t));
hipMemset(d_MAT_IDX61e04decd970, 0, sizeof(uint64_t));
auto MAT61e04decd970aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT61e04decd970);
DBDecimalType* d_MAT61e04decd970aggr0__tmp_attr0;
hipMalloc(&d_MAT61e04decd970aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT61e04decd970);
main_61e04de92310<<<std::ceil((float)COUNT61e04debb940/32.), 32>>>(COUNT61e04debb940, d_MAT61e04decd970aggr0__tmp_attr0, d_MAT_IDX61e04decd970, d_aggr0__tmp_attr0);
hipFree(d_MAT_IDX61e04decd970);
hipMemcpy(MAT61e04decd970aggr0__tmp_attr0, d_MAT61e04decd970aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT61e04decd970, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT61e04decd970; i++) { std::cout << MAT61e04decd970aggr0__tmp_attr0[i] << "\t";
std::cout << std::endl; }
}