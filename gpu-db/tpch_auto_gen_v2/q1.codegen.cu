#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
template<typename HASHTABLE_INSERT>
__global__ void count_56fe32df52c0(HASHTABLE_INSERT HT_56fe32ea4280, DBCharType* lineitem__l_linestatus, DBCharType* lineitem__l_returnflag, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 10471, Predicate::lte))) return;
uint64_t KEY_56fe32ea4280 = 0;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];

KEY_56fe32ea4280 |= reg_lineitem__l_returnflag;
auto reg_lineitem__l_linestatus = lineitem__l_linestatus[tid];
KEY_56fe32ea4280 <<= 8;
KEY_56fe32ea4280 |= reg_lineitem__l_linestatus;
//Create aggregation hash table
HT_56fe32ea4280.insert(cuco::pair{KEY_56fe32ea4280, 1});
}
template<typename HASHTABLE_FIND>
__global__ void main_56fe32df52c0(HASHTABLE_FIND HT_56fe32ea4280, DBCharType* KEY_56fe32ea4280lineitem__l_linestatus, DBCharType* KEY_56fe32ea4280lineitem__l_returnflag, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr0__tmp_attr1, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, DBI64Type* aggr0__tmp_attr9, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, DBDecimalType* aggr_rw__rw2, DBI64Type* aggr_rw__rw3, DBDecimalType* aggr_rw__rw4, DBI64Type* aggr_rw__rw5, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBCharType* lineitem__l_linestatus, DBDecimalType* lineitem__l_quantity, DBCharType* lineitem__l_returnflag, DBDateType* lineitem__l_shipdate, DBDecimalType* lineitem__l_tax, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 10471, Predicate::lte))) return;
uint64_t KEY_56fe32ea4280 = 0;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];

KEY_56fe32ea4280 |= reg_lineitem__l_returnflag;
auto reg_lineitem__l_linestatus = lineitem__l_linestatus[tid];
KEY_56fe32ea4280 <<= 8;
KEY_56fe32ea4280 |= reg_lineitem__l_linestatus;
//Aggregate in hashtable
auto buf_idx_56fe32ea4280 = HT_56fe32ea4280.find(KEY_56fe32ea4280)->second;
aggregate_sum(&aggr0__tmp_attr9[buf_idx_56fe32ea4280], 1);
auto reg_lineitem__l_tax = lineitem__l_tax[tid];
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr5 = ((reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount))) * ((1) + (reg_lineitem__l_tax));
aggregate_sum(&aggr0__tmp_attr4[buf_idx_56fe32ea4280], reg_map0__tmp_attr5);
auto reg_map0__tmp_attr3 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_56fe32ea4280], reg_map0__tmp_attr3);
aggregate_sum(&aggr0__tmp_attr1[buf_idx_56fe32ea4280], reg_lineitem__l_extendedprice);
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_56fe32ea4280], reg_lineitem__l_quantity);
aggregate_sum(&aggr_rw__rw0[buf_idx_56fe32ea4280], reg_lineitem__l_discount);
aggregate_sum(&aggr_rw__rw1[buf_idx_56fe32ea4280], 1);
aggregate_sum(&aggr_rw__rw2[buf_idx_56fe32ea4280], reg_lineitem__l_extendedprice);
aggregate_sum(&aggr_rw__rw3[buf_idx_56fe32ea4280], 1);
aggregate_sum(&aggr_rw__rw4[buf_idx_56fe32ea4280], reg_lineitem__l_quantity);
aggregate_sum(&aggr_rw__rw5[buf_idx_56fe32ea4280], 1);
KEY_56fe32ea4280lineitem__l_returnflag[buf_idx_56fe32ea4280] = reg_lineitem__l_returnflag;
KEY_56fe32ea4280lineitem__l_linestatus[buf_idx_56fe32ea4280] = reg_lineitem__l_linestatus;
}
__global__ void count_56fe32e785a0(uint64_t* COUNT56fe32e85170, size_t COUNT56fe32ea4280) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT56fe32ea4280) return;
//Materialize count
atomicAdd((int*)COUNT56fe32e85170, 1);
}
__global__ void main_56fe32e785a0(size_t COUNT56fe32ea4280, DBDecimalType* MAT56fe32e85170aggr0__tmp_attr0, DBDecimalType* MAT56fe32e85170aggr0__tmp_attr1, DBDecimalType* MAT56fe32e85170aggr0__tmp_attr2, DBDecimalType* MAT56fe32e85170aggr0__tmp_attr4, DBDecimalType* MAT56fe32e85170aggr0__tmp_attr6, DBDecimalType* MAT56fe32e85170aggr0__tmp_attr7, DBDecimalType* MAT56fe32e85170aggr0__tmp_attr8, DBI64Type* MAT56fe32e85170aggr0__tmp_attr9, DBCharType* MAT56fe32e85170lineitem__l_linestatus, DBCharType* MAT56fe32e85170lineitem__l_returnflag, uint64_t* MAT_IDX56fe32e85170, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr0__tmp_attr1, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, DBI64Type* aggr0__tmp_attr9, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, DBDecimalType* aggr_rw__rw2, DBI64Type* aggr_rw__rw3, DBDecimalType* aggr_rw__rw4, DBI64Type* aggr_rw__rw5, DBCharType* lineitem__l_linestatus, DBCharType* lineitem__l_returnflag) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT56fe32ea4280) return;
//Materialize buffers
auto mat_idx56fe32e85170 = atomicAdd((int*)MAT_IDX56fe32e85170, 1);
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];
MAT56fe32e85170lineitem__l_returnflag[mat_idx56fe32e85170] = reg_lineitem__l_returnflag;
auto reg_lineitem__l_linestatus = lineitem__l_linestatus[tid];
MAT56fe32e85170lineitem__l_linestatus[mat_idx56fe32e85170] = reg_lineitem__l_linestatus;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT56fe32e85170aggr0__tmp_attr0[mat_idx56fe32e85170] = reg_aggr0__tmp_attr0;
auto reg_aggr0__tmp_attr1 = aggr0__tmp_attr1[tid];
MAT56fe32e85170aggr0__tmp_attr1[mat_idx56fe32e85170] = reg_aggr0__tmp_attr1;
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT56fe32e85170aggr0__tmp_attr2[mat_idx56fe32e85170] = reg_aggr0__tmp_attr2;
auto reg_aggr0__tmp_attr4 = aggr0__tmp_attr4[tid];
MAT56fe32e85170aggr0__tmp_attr4[mat_idx56fe32e85170] = reg_aggr0__tmp_attr4;
auto reg_aggr_rw__rw5 = aggr_rw__rw5[tid];
auto reg_aggr_rw__rw4 = aggr_rw__rw4[tid];
auto reg_aggr0__tmp_attr6 = (reg_aggr_rw__rw4) / ((DBDecimalType)(reg_aggr_rw__rw5));
MAT56fe32e85170aggr0__tmp_attr6[mat_idx56fe32e85170] = reg_aggr0__tmp_attr6;
auto reg_aggr_rw__rw3 = aggr_rw__rw3[tid];
auto reg_aggr_rw__rw2 = aggr_rw__rw2[tid];
auto reg_aggr0__tmp_attr7 = (reg_aggr_rw__rw2) / ((DBDecimalType)(reg_aggr_rw__rw3));
MAT56fe32e85170aggr0__tmp_attr7[mat_idx56fe32e85170] = reg_aggr0__tmp_attr7;
auto reg_aggr_rw__rw1 = aggr_rw__rw1[tid];
auto reg_aggr_rw__rw0 = aggr_rw__rw0[tid];
auto reg_aggr0__tmp_attr8 = (reg_aggr_rw__rw0) / ((DBDecimalType)(reg_aggr_rw__rw1));
MAT56fe32e85170aggr0__tmp_attr8[mat_idx56fe32e85170] = reg_aggr0__tmp_attr8;
auto reg_aggr0__tmp_attr9 = aggr0__tmp_attr9[tid];
MAT56fe32e85170aggr0__tmp_attr9[mat_idx56fe32e85170] = reg_aggr0__tmp_attr9;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Create aggregation hash table
auto d_HT_56fe32ea4280 = cuco::static_map{ (int)5930889*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_56fe32df52c0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_HT_56fe32ea4280.ref(cuco::insert), d_lineitem__l_linestatus, d_lineitem__l_returnflag, d_lineitem__l_shipdate, lineitem_size);
size_t COUNT56fe32ea4280 = d_HT_56fe32ea4280.size();
thrust::device_vector<int64_t> keys_56fe32ea4280(COUNT56fe32ea4280), vals_56fe32ea4280(COUNT56fe32ea4280);
d_HT_56fe32ea4280.retrieve_all(keys_56fe32ea4280.begin(), vals_56fe32ea4280.begin());
thrust::host_vector<int64_t> h_keys_56fe32ea4280(COUNT56fe32ea4280);
thrust::copy(keys_56fe32ea4280.begin(), keys_56fe32ea4280.end(), h_keys_56fe32ea4280.begin());
thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_56fe32ea4280(COUNT56fe32ea4280);
for (int i=0; i < COUNT56fe32ea4280; i++)
{actual_dict_56fe32ea4280[i] = cuco::make_pair(h_keys_56fe32ea4280[i], i);}
d_HT_56fe32ea4280.clear();
d_HT_56fe32ea4280.insert(actual_dict_56fe32ea4280.begin(), actual_dict_56fe32ea4280.end());
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr9;
hipMalloc(&d_aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT56fe32ea4280);
hipMemset(d_aggr0__tmp_attr9, 0, sizeof(DBI64Type) * COUNT56fe32ea4280);
DBDecimalType* d_aggr0__tmp_attr4;
hipMalloc(&d_aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT56fe32ea4280);
hipMemset(d_aggr0__tmp_attr4, 0, sizeof(DBDecimalType) * COUNT56fe32ea4280);
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT56fe32ea4280);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT56fe32ea4280);
DBDecimalType* d_aggr0__tmp_attr1;
hipMalloc(&d_aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT56fe32ea4280);
hipMemset(d_aggr0__tmp_attr1, 0, sizeof(DBDecimalType) * COUNT56fe32ea4280);
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT56fe32ea4280);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT56fe32ea4280);
DBDecimalType* d_aggr_rw__rw0;
hipMalloc(&d_aggr_rw__rw0, sizeof(DBDecimalType) * COUNT56fe32ea4280);
hipMemset(d_aggr_rw__rw0, 0, sizeof(DBDecimalType) * COUNT56fe32ea4280);
DBI64Type* d_aggr_rw__rw1;
hipMalloc(&d_aggr_rw__rw1, sizeof(DBI64Type) * COUNT56fe32ea4280);
hipMemset(d_aggr_rw__rw1, 0, sizeof(DBI64Type) * COUNT56fe32ea4280);
DBDecimalType* d_aggr_rw__rw2;
hipMalloc(&d_aggr_rw__rw2, sizeof(DBDecimalType) * COUNT56fe32ea4280);
hipMemset(d_aggr_rw__rw2, 0, sizeof(DBDecimalType) * COUNT56fe32ea4280);
DBI64Type* d_aggr_rw__rw3;
hipMalloc(&d_aggr_rw__rw3, sizeof(DBI64Type) * COUNT56fe32ea4280);
hipMemset(d_aggr_rw__rw3, 0, sizeof(DBI64Type) * COUNT56fe32ea4280);
DBDecimalType* d_aggr_rw__rw4;
hipMalloc(&d_aggr_rw__rw4, sizeof(DBDecimalType) * COUNT56fe32ea4280);
hipMemset(d_aggr_rw__rw4, 0, sizeof(DBDecimalType) * COUNT56fe32ea4280);
DBI64Type* d_aggr_rw__rw5;
hipMalloc(&d_aggr_rw__rw5, sizeof(DBI64Type) * COUNT56fe32ea4280);
hipMemset(d_aggr_rw__rw5, 0, sizeof(DBI64Type) * COUNT56fe32ea4280);
DBCharType* d_KEY_56fe32ea4280lineitem__l_returnflag;
hipMalloc(&d_KEY_56fe32ea4280lineitem__l_returnflag, sizeof(DBCharType) * COUNT56fe32ea4280);
hipMemset(d_KEY_56fe32ea4280lineitem__l_returnflag, 0, sizeof(DBCharType) * COUNT56fe32ea4280);
DBCharType* d_KEY_56fe32ea4280lineitem__l_linestatus;
hipMalloc(&d_KEY_56fe32ea4280lineitem__l_linestatus, sizeof(DBCharType) * COUNT56fe32ea4280);
hipMemset(d_KEY_56fe32ea4280lineitem__l_linestatus, 0, sizeof(DBCharType) * COUNT56fe32ea4280);
main_56fe32df52c0<<<std::ceil((float)lineitem_size/32.), 32>>>(d_HT_56fe32ea4280.ref(cuco::find), d_KEY_56fe32ea4280lineitem__l_linestatus, d_KEY_56fe32ea4280lineitem__l_returnflag, d_aggr0__tmp_attr0, d_aggr0__tmp_attr1, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_aggr0__tmp_attr9, d_aggr_rw__rw0, d_aggr_rw__rw1, d_aggr_rw__rw2, d_aggr_rw__rw3, d_aggr_rw__rw4, d_aggr_rw__rw5, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_linestatus, d_lineitem__l_quantity, d_lineitem__l_returnflag, d_lineitem__l_shipdate, d_lineitem__l_tax, lineitem_size);
//Materialize count
uint64_t* d_COUNT56fe32e85170;
hipMalloc(&d_COUNT56fe32e85170, sizeof(uint64_t));
hipMemset(d_COUNT56fe32e85170, 0, sizeof(uint64_t));
count_56fe32e785a0<<<std::ceil((float)COUNT56fe32ea4280/32.), 32>>>(d_COUNT56fe32e85170, COUNT56fe32ea4280);
uint64_t COUNT56fe32e85170;
hipMemcpy(&COUNT56fe32e85170, d_COUNT56fe32e85170, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT56fe32e85170);
//Materialize buffers
uint64_t* d_MAT_IDX56fe32e85170;
hipMalloc(&d_MAT_IDX56fe32e85170, sizeof(uint64_t));
hipMemset(d_MAT_IDX56fe32e85170, 0, sizeof(uint64_t));
auto MAT56fe32e85170lineitem__l_returnflag = (DBCharType*)malloc(sizeof(DBCharType) * COUNT56fe32e85170);
DBCharType* d_MAT56fe32e85170lineitem__l_returnflag;
hipMalloc(&d_MAT56fe32e85170lineitem__l_returnflag, sizeof(DBCharType) * COUNT56fe32e85170);
auto MAT56fe32e85170lineitem__l_linestatus = (DBCharType*)malloc(sizeof(DBCharType) * COUNT56fe32e85170);
DBCharType* d_MAT56fe32e85170lineitem__l_linestatus;
hipMalloc(&d_MAT56fe32e85170lineitem__l_linestatus, sizeof(DBCharType) * COUNT56fe32e85170);
auto MAT56fe32e85170aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT56fe32e85170);
DBDecimalType* d_MAT56fe32e85170aggr0__tmp_attr0;
hipMalloc(&d_MAT56fe32e85170aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT56fe32e85170);
auto MAT56fe32e85170aggr0__tmp_attr1 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT56fe32e85170);
DBDecimalType* d_MAT56fe32e85170aggr0__tmp_attr1;
hipMalloc(&d_MAT56fe32e85170aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT56fe32e85170);
auto MAT56fe32e85170aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT56fe32e85170);
DBDecimalType* d_MAT56fe32e85170aggr0__tmp_attr2;
hipMalloc(&d_MAT56fe32e85170aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT56fe32e85170);
auto MAT56fe32e85170aggr0__tmp_attr4 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT56fe32e85170);
DBDecimalType* d_MAT56fe32e85170aggr0__tmp_attr4;
hipMalloc(&d_MAT56fe32e85170aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT56fe32e85170);
auto MAT56fe32e85170aggr0__tmp_attr6 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT56fe32e85170);
DBDecimalType* d_MAT56fe32e85170aggr0__tmp_attr6;
hipMalloc(&d_MAT56fe32e85170aggr0__tmp_attr6, sizeof(DBDecimalType) * COUNT56fe32e85170);
auto MAT56fe32e85170aggr0__tmp_attr7 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT56fe32e85170);
DBDecimalType* d_MAT56fe32e85170aggr0__tmp_attr7;
hipMalloc(&d_MAT56fe32e85170aggr0__tmp_attr7, sizeof(DBDecimalType) * COUNT56fe32e85170);
auto MAT56fe32e85170aggr0__tmp_attr8 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT56fe32e85170);
DBDecimalType* d_MAT56fe32e85170aggr0__tmp_attr8;
hipMalloc(&d_MAT56fe32e85170aggr0__tmp_attr8, sizeof(DBDecimalType) * COUNT56fe32e85170);
auto MAT56fe32e85170aggr0__tmp_attr9 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT56fe32e85170);
DBI64Type* d_MAT56fe32e85170aggr0__tmp_attr9;
hipMalloc(&d_MAT56fe32e85170aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT56fe32e85170);
main_56fe32e785a0<<<std::ceil((float)COUNT56fe32ea4280/32.), 32>>>(COUNT56fe32ea4280, d_MAT56fe32e85170aggr0__tmp_attr0, d_MAT56fe32e85170aggr0__tmp_attr1, d_MAT56fe32e85170aggr0__tmp_attr2, d_MAT56fe32e85170aggr0__tmp_attr4, d_MAT56fe32e85170aggr0__tmp_attr6, d_MAT56fe32e85170aggr0__tmp_attr7, d_MAT56fe32e85170aggr0__tmp_attr8, d_MAT56fe32e85170aggr0__tmp_attr9, d_MAT56fe32e85170lineitem__l_linestatus, d_MAT56fe32e85170lineitem__l_returnflag, d_MAT_IDX56fe32e85170, d_aggr0__tmp_attr0, d_aggr0__tmp_attr1, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_aggr0__tmp_attr9, d_aggr_rw__rw0, d_aggr_rw__rw1, d_aggr_rw__rw2, d_aggr_rw__rw3, d_aggr_rw__rw4, d_aggr_rw__rw5, d_KEY_56fe32ea4280lineitem__l_linestatus, d_KEY_56fe32ea4280lineitem__l_returnflag);
hipFree(d_MAT_IDX56fe32e85170);
hipMemcpy(MAT56fe32e85170lineitem__l_returnflag, d_MAT56fe32e85170lineitem__l_returnflag, sizeof(DBCharType) * COUNT56fe32e85170, hipMemcpyDeviceToHost);
hipMemcpy(MAT56fe32e85170lineitem__l_linestatus, d_MAT56fe32e85170lineitem__l_linestatus, sizeof(DBCharType) * COUNT56fe32e85170, hipMemcpyDeviceToHost);
hipMemcpy(MAT56fe32e85170aggr0__tmp_attr0, d_MAT56fe32e85170aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT56fe32e85170, hipMemcpyDeviceToHost);
hipMemcpy(MAT56fe32e85170aggr0__tmp_attr1, d_MAT56fe32e85170aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT56fe32e85170, hipMemcpyDeviceToHost);
hipMemcpy(MAT56fe32e85170aggr0__tmp_attr2, d_MAT56fe32e85170aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT56fe32e85170, hipMemcpyDeviceToHost);
hipMemcpy(MAT56fe32e85170aggr0__tmp_attr4, d_MAT56fe32e85170aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT56fe32e85170, hipMemcpyDeviceToHost);
hipMemcpy(MAT56fe32e85170aggr0__tmp_attr6, d_MAT56fe32e85170aggr0__tmp_attr6, sizeof(DBDecimalType) * COUNT56fe32e85170, hipMemcpyDeviceToHost);
hipMemcpy(MAT56fe32e85170aggr0__tmp_attr7, d_MAT56fe32e85170aggr0__tmp_attr7, sizeof(DBDecimalType) * COUNT56fe32e85170, hipMemcpyDeviceToHost);
hipMemcpy(MAT56fe32e85170aggr0__tmp_attr8, d_MAT56fe32e85170aggr0__tmp_attr8, sizeof(DBDecimalType) * COUNT56fe32e85170, hipMemcpyDeviceToHost);
hipMemcpy(MAT56fe32e85170aggr0__tmp_attr9, d_MAT56fe32e85170aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT56fe32e85170, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT56fe32e85170; i++) { std::cout << MAT56fe32e85170lineitem__l_returnflag[i] << "\t";
std::cout << MAT56fe32e85170lineitem__l_linestatus[i] << "\t";
std::cout << MAT56fe32e85170aggr0__tmp_attr0[i] << "\t";
std::cout << MAT56fe32e85170aggr0__tmp_attr1[i] << "\t";
std::cout << MAT56fe32e85170aggr0__tmp_attr2[i] << "\t";
std::cout << MAT56fe32e85170aggr0__tmp_attr4[i] << "\t";
std::cout << MAT56fe32e85170aggr0__tmp_attr6[i] << "\t";
std::cout << MAT56fe32e85170aggr0__tmp_attr7[i] << "\t";
std::cout << MAT56fe32e85170aggr0__tmp_attr8[i] << "\t";
std::cout << MAT56fe32e85170aggr0__tmp_attr9[i] << "\t";
std::cout << std::endl; }
}