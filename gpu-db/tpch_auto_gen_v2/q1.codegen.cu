#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
template<typename HASHTABLE_INSERT>
__global__ void count_57c908d41c20(HASHTABLE_INSERT HT_57c908d0d780, DBCharType* lineitem__l_linestatus, DBCharType* lineitem__l_returnflag, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 10471, Predicate::lte))) return;
uint64_t KEY_57c908d0d780 = 0;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];

KEY_57c908d0d780 |= reg_lineitem__l_returnflag;
auto reg_lineitem__l_linestatus = lineitem__l_linestatus[tid];
KEY_57c908d0d780 <<= 8;
KEY_57c908d0d780 |= reg_lineitem__l_linestatus;
//Create aggregation hash table
HT_57c908d0d780.insert(cuco::pair{KEY_57c908d0d780, 1});
}
template<typename HASHTABLE_FIND>
__global__ void main_57c908d41c20(HASHTABLE_FIND HT_57c908d0d780, DBCharType* KEY_57c908d0d780lineitem__l_linestatus, DBCharType* KEY_57c908d0d780lineitem__l_returnflag, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr0__tmp_attr1, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, DBI64Type* aggr0__tmp_attr9, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, DBDecimalType* aggr_rw__rw2, DBI64Type* aggr_rw__rw3, DBDecimalType* aggr_rw__rw4, DBI64Type* aggr_rw__rw5, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBCharType* lineitem__l_linestatus, DBDecimalType* lineitem__l_quantity, DBCharType* lineitem__l_returnflag, DBDateType* lineitem__l_shipdate, DBDecimalType* lineitem__l_tax, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 10471, Predicate::lte))) return;
uint64_t KEY_57c908d0d780 = 0;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];

KEY_57c908d0d780 |= reg_lineitem__l_returnflag;
auto reg_lineitem__l_linestatus = lineitem__l_linestatus[tid];
KEY_57c908d0d780 <<= 8;
KEY_57c908d0d780 |= reg_lineitem__l_linestatus;
//Aggregate in hashtable
auto buf_idx_57c908d0d780 = HT_57c908d0d780.find(KEY_57c908d0d780)->second;
aggregate_sum(&aggr0__tmp_attr9[buf_idx_57c908d0d780], 1);
auto reg_lineitem__l_tax = lineitem__l_tax[tid];
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr5 = ((reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount))) * ((1) + (reg_lineitem__l_tax));
aggregate_sum(&aggr0__tmp_attr4[buf_idx_57c908d0d780], reg_map0__tmp_attr5);
auto reg_map0__tmp_attr3 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr2[buf_idx_57c908d0d780], reg_map0__tmp_attr3);
aggregate_sum(&aggr0__tmp_attr1[buf_idx_57c908d0d780], reg_lineitem__l_extendedprice);
auto reg_lineitem__l_quantity = lineitem__l_quantity[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_57c908d0d780], reg_lineitem__l_quantity);
aggregate_sum(&aggr_rw__rw0[buf_idx_57c908d0d780], reg_lineitem__l_discount);
aggregate_sum(&aggr_rw__rw1[buf_idx_57c908d0d780], 1);
aggregate_sum(&aggr_rw__rw2[buf_idx_57c908d0d780], reg_lineitem__l_extendedprice);
aggregate_sum(&aggr_rw__rw3[buf_idx_57c908d0d780], 1);
aggregate_sum(&aggr_rw__rw4[buf_idx_57c908d0d780], reg_lineitem__l_quantity);
aggregate_sum(&aggr_rw__rw5[buf_idx_57c908d0d780], 1);
KEY_57c908d0d780lineitem__l_returnflag[buf_idx_57c908d0d780] = reg_lineitem__l_returnflag;
KEY_57c908d0d780lineitem__l_linestatus[buf_idx_57c908d0d780] = reg_lineitem__l_linestatus;
}
__global__ void count_57c908d41e60(uint64_t* COUNT57c908cee9c0, size_t COUNT57c908d0d780) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT57c908d0d780) return;
//Materialize count
atomicAdd((int*)COUNT57c908cee9c0, 1);
}
__global__ void main_57c908d41e60(size_t COUNT57c908d0d780, DBDecimalType* MAT57c908cee9c0aggr0__tmp_attr0, DBDecimalType* MAT57c908cee9c0aggr0__tmp_attr1, DBDecimalType* MAT57c908cee9c0aggr0__tmp_attr2, DBDecimalType* MAT57c908cee9c0aggr0__tmp_attr4, DBDecimalType* MAT57c908cee9c0aggr0__tmp_attr6, DBDecimalType* MAT57c908cee9c0aggr0__tmp_attr7, DBDecimalType* MAT57c908cee9c0aggr0__tmp_attr8, DBI64Type* MAT57c908cee9c0aggr0__tmp_attr9, DBCharType* MAT57c908cee9c0lineitem__l_linestatus, DBCharType* MAT57c908cee9c0lineitem__l_returnflag, uint64_t* MAT_IDX57c908cee9c0, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr0__tmp_attr1, DBDecimalType* aggr0__tmp_attr2, DBDecimalType* aggr0__tmp_attr4, DBI64Type* aggr0__tmp_attr9, DBDecimalType* aggr_rw__rw0, DBI64Type* aggr_rw__rw1, DBDecimalType* aggr_rw__rw2, DBI64Type* aggr_rw__rw3, DBDecimalType* aggr_rw__rw4, DBI64Type* aggr_rw__rw5, DBCharType* lineitem__l_linestatus, DBCharType* lineitem__l_returnflag) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT57c908d0d780) return;
//Materialize buffers
auto mat_idx57c908cee9c0 = atomicAdd((int*)MAT_IDX57c908cee9c0, 1);
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];
MAT57c908cee9c0lineitem__l_returnflag[mat_idx57c908cee9c0] = reg_lineitem__l_returnflag;
auto reg_lineitem__l_linestatus = lineitem__l_linestatus[tid];
MAT57c908cee9c0lineitem__l_linestatus[mat_idx57c908cee9c0] = reg_lineitem__l_linestatus;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT57c908cee9c0aggr0__tmp_attr0[mat_idx57c908cee9c0] = reg_aggr0__tmp_attr0;
auto reg_aggr0__tmp_attr1 = aggr0__tmp_attr1[tid];
MAT57c908cee9c0aggr0__tmp_attr1[mat_idx57c908cee9c0] = reg_aggr0__tmp_attr1;
auto reg_aggr0__tmp_attr2 = aggr0__tmp_attr2[tid];
MAT57c908cee9c0aggr0__tmp_attr2[mat_idx57c908cee9c0] = reg_aggr0__tmp_attr2;
auto reg_aggr0__tmp_attr4 = aggr0__tmp_attr4[tid];
MAT57c908cee9c0aggr0__tmp_attr4[mat_idx57c908cee9c0] = reg_aggr0__tmp_attr4;
auto reg_aggr_rw__rw5 = aggr_rw__rw5[tid];
auto reg_aggr_rw__rw4 = aggr_rw__rw4[tid];
auto reg_aggr0__tmp_attr6 = (reg_aggr_rw__rw4) / ((DBDecimalType)(reg_aggr_rw__rw5));
MAT57c908cee9c0aggr0__tmp_attr6[mat_idx57c908cee9c0] = reg_aggr0__tmp_attr6;
auto reg_aggr_rw__rw3 = aggr_rw__rw3[tid];
auto reg_aggr_rw__rw2 = aggr_rw__rw2[tid];
auto reg_aggr0__tmp_attr7 = (reg_aggr_rw__rw2) / ((DBDecimalType)(reg_aggr_rw__rw3));
MAT57c908cee9c0aggr0__tmp_attr7[mat_idx57c908cee9c0] = reg_aggr0__tmp_attr7;
auto reg_aggr_rw__rw1 = aggr_rw__rw1[tid];
auto reg_aggr_rw__rw0 = aggr_rw__rw0[tid];
auto reg_aggr0__tmp_attr8 = (reg_aggr_rw__rw0) / ((DBDecimalType)(reg_aggr_rw__rw1));
MAT57c908cee9c0aggr0__tmp_attr8[mat_idx57c908cee9c0] = reg_aggr0__tmp_attr8;
auto reg_aggr0__tmp_attr9 = aggr0__tmp_attr9[tid];
MAT57c908cee9c0aggr0__tmp_attr9[mat_idx57c908cee9c0] = reg_aggr0__tmp_attr9;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Create aggregation hash table
auto d_HT_57c908d0d780 = cuco::static_map{ (int)5930889*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_57c908d41c20<<<std::ceil((float)lineitem_size/32.), 32>>>(d_HT_57c908d0d780.ref(cuco::insert), d_lineitem__l_linestatus, d_lineitem__l_returnflag, d_lineitem__l_shipdate, lineitem_size);
size_t COUNT57c908d0d780 = d_HT_57c908d0d780.size();
thrust::device_vector<int64_t> keys_57c908d0d780(COUNT57c908d0d780), vals_57c908d0d780(COUNT57c908d0d780);
d_HT_57c908d0d780.retrieve_all(keys_57c908d0d780.begin(), vals_57c908d0d780.begin());
d_HT_57c908d0d780.clear();
int64_t* raw_keys57c908d0d780 = thrust::raw_pointer_cast(keys_57c908d0d780.data());
insertKeys<<<std::ceil((float)COUNT57c908d0d780/32.), 32>>>(raw_keys57c908d0d780, d_HT_57c908d0d780.ref(cuco::insert), COUNT57c908d0d780);
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr9;
hipMalloc(&d_aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT57c908d0d780);
hipMemset(d_aggr0__tmp_attr9, 0, sizeof(DBI64Type) * COUNT57c908d0d780);
DBDecimalType* d_aggr0__tmp_attr4;
hipMalloc(&d_aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT57c908d0d780);
hipMemset(d_aggr0__tmp_attr4, 0, sizeof(DBDecimalType) * COUNT57c908d0d780);
DBDecimalType* d_aggr0__tmp_attr2;
hipMalloc(&d_aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT57c908d0d780);
hipMemset(d_aggr0__tmp_attr2, 0, sizeof(DBDecimalType) * COUNT57c908d0d780);
DBDecimalType* d_aggr0__tmp_attr1;
hipMalloc(&d_aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT57c908d0d780);
hipMemset(d_aggr0__tmp_attr1, 0, sizeof(DBDecimalType) * COUNT57c908d0d780);
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT57c908d0d780);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT57c908d0d780);
DBDecimalType* d_aggr_rw__rw0;
hipMalloc(&d_aggr_rw__rw0, sizeof(DBDecimalType) * COUNT57c908d0d780);
hipMemset(d_aggr_rw__rw0, 0, sizeof(DBDecimalType) * COUNT57c908d0d780);
DBI64Type* d_aggr_rw__rw1;
hipMalloc(&d_aggr_rw__rw1, sizeof(DBI64Type) * COUNT57c908d0d780);
hipMemset(d_aggr_rw__rw1, 0, sizeof(DBI64Type) * COUNT57c908d0d780);
DBDecimalType* d_aggr_rw__rw2;
hipMalloc(&d_aggr_rw__rw2, sizeof(DBDecimalType) * COUNT57c908d0d780);
hipMemset(d_aggr_rw__rw2, 0, sizeof(DBDecimalType) * COUNT57c908d0d780);
DBI64Type* d_aggr_rw__rw3;
hipMalloc(&d_aggr_rw__rw3, sizeof(DBI64Type) * COUNT57c908d0d780);
hipMemset(d_aggr_rw__rw3, 0, sizeof(DBI64Type) * COUNT57c908d0d780);
DBDecimalType* d_aggr_rw__rw4;
hipMalloc(&d_aggr_rw__rw4, sizeof(DBDecimalType) * COUNT57c908d0d780);
hipMemset(d_aggr_rw__rw4, 0, sizeof(DBDecimalType) * COUNT57c908d0d780);
DBI64Type* d_aggr_rw__rw5;
hipMalloc(&d_aggr_rw__rw5, sizeof(DBI64Type) * COUNT57c908d0d780);
hipMemset(d_aggr_rw__rw5, 0, sizeof(DBI64Type) * COUNT57c908d0d780);
DBCharType* d_KEY_57c908d0d780lineitem__l_returnflag;
hipMalloc(&d_KEY_57c908d0d780lineitem__l_returnflag, sizeof(DBCharType) * COUNT57c908d0d780);
hipMemset(d_KEY_57c908d0d780lineitem__l_returnflag, 0, sizeof(DBCharType) * COUNT57c908d0d780);
DBCharType* d_KEY_57c908d0d780lineitem__l_linestatus;
hipMalloc(&d_KEY_57c908d0d780lineitem__l_linestatus, sizeof(DBCharType) * COUNT57c908d0d780);
hipMemset(d_KEY_57c908d0d780lineitem__l_linestatus, 0, sizeof(DBCharType) * COUNT57c908d0d780);
main_57c908d41c20<<<std::ceil((float)lineitem_size/32.), 32>>>(d_HT_57c908d0d780.ref(cuco::find), d_KEY_57c908d0d780lineitem__l_linestatus, d_KEY_57c908d0d780lineitem__l_returnflag, d_aggr0__tmp_attr0, d_aggr0__tmp_attr1, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_aggr0__tmp_attr9, d_aggr_rw__rw0, d_aggr_rw__rw1, d_aggr_rw__rw2, d_aggr_rw__rw3, d_aggr_rw__rw4, d_aggr_rw__rw5, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_linestatus, d_lineitem__l_quantity, d_lineitem__l_returnflag, d_lineitem__l_shipdate, d_lineitem__l_tax, lineitem_size);
//Materialize count
uint64_t* d_COUNT57c908cee9c0;
hipMalloc(&d_COUNT57c908cee9c0, sizeof(uint64_t));
hipMemset(d_COUNT57c908cee9c0, 0, sizeof(uint64_t));
count_57c908d41e60<<<std::ceil((float)COUNT57c908d0d780/32.), 32>>>(d_COUNT57c908cee9c0, COUNT57c908d0d780);
uint64_t COUNT57c908cee9c0;
hipMemcpy(&COUNT57c908cee9c0, d_COUNT57c908cee9c0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX57c908cee9c0;
hipMalloc(&d_MAT_IDX57c908cee9c0, sizeof(uint64_t));
hipMemset(d_MAT_IDX57c908cee9c0, 0, sizeof(uint64_t));
auto MAT57c908cee9c0lineitem__l_returnflag = (DBCharType*)malloc(sizeof(DBCharType) * COUNT57c908cee9c0);
DBCharType* d_MAT57c908cee9c0lineitem__l_returnflag;
hipMalloc(&d_MAT57c908cee9c0lineitem__l_returnflag, sizeof(DBCharType) * COUNT57c908cee9c0);
auto MAT57c908cee9c0lineitem__l_linestatus = (DBCharType*)malloc(sizeof(DBCharType) * COUNT57c908cee9c0);
DBCharType* d_MAT57c908cee9c0lineitem__l_linestatus;
hipMalloc(&d_MAT57c908cee9c0lineitem__l_linestatus, sizeof(DBCharType) * COUNT57c908cee9c0);
auto MAT57c908cee9c0aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT57c908cee9c0);
DBDecimalType* d_MAT57c908cee9c0aggr0__tmp_attr0;
hipMalloc(&d_MAT57c908cee9c0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT57c908cee9c0);
auto MAT57c908cee9c0aggr0__tmp_attr1 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT57c908cee9c0);
DBDecimalType* d_MAT57c908cee9c0aggr0__tmp_attr1;
hipMalloc(&d_MAT57c908cee9c0aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT57c908cee9c0);
auto MAT57c908cee9c0aggr0__tmp_attr2 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT57c908cee9c0);
DBDecimalType* d_MAT57c908cee9c0aggr0__tmp_attr2;
hipMalloc(&d_MAT57c908cee9c0aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT57c908cee9c0);
auto MAT57c908cee9c0aggr0__tmp_attr4 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT57c908cee9c0);
DBDecimalType* d_MAT57c908cee9c0aggr0__tmp_attr4;
hipMalloc(&d_MAT57c908cee9c0aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT57c908cee9c0);
auto MAT57c908cee9c0aggr0__tmp_attr6 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT57c908cee9c0);
DBDecimalType* d_MAT57c908cee9c0aggr0__tmp_attr6;
hipMalloc(&d_MAT57c908cee9c0aggr0__tmp_attr6, sizeof(DBDecimalType) * COUNT57c908cee9c0);
auto MAT57c908cee9c0aggr0__tmp_attr7 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT57c908cee9c0);
DBDecimalType* d_MAT57c908cee9c0aggr0__tmp_attr7;
hipMalloc(&d_MAT57c908cee9c0aggr0__tmp_attr7, sizeof(DBDecimalType) * COUNT57c908cee9c0);
auto MAT57c908cee9c0aggr0__tmp_attr8 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT57c908cee9c0);
DBDecimalType* d_MAT57c908cee9c0aggr0__tmp_attr8;
hipMalloc(&d_MAT57c908cee9c0aggr0__tmp_attr8, sizeof(DBDecimalType) * COUNT57c908cee9c0);
auto MAT57c908cee9c0aggr0__tmp_attr9 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT57c908cee9c0);
DBI64Type* d_MAT57c908cee9c0aggr0__tmp_attr9;
hipMalloc(&d_MAT57c908cee9c0aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT57c908cee9c0);
main_57c908d41e60<<<std::ceil((float)COUNT57c908d0d780/32.), 32>>>(COUNT57c908d0d780, d_MAT57c908cee9c0aggr0__tmp_attr0, d_MAT57c908cee9c0aggr0__tmp_attr1, d_MAT57c908cee9c0aggr0__tmp_attr2, d_MAT57c908cee9c0aggr0__tmp_attr4, d_MAT57c908cee9c0aggr0__tmp_attr6, d_MAT57c908cee9c0aggr0__tmp_attr7, d_MAT57c908cee9c0aggr0__tmp_attr8, d_MAT57c908cee9c0aggr0__tmp_attr9, d_MAT57c908cee9c0lineitem__l_linestatus, d_MAT57c908cee9c0lineitem__l_returnflag, d_MAT_IDX57c908cee9c0, d_aggr0__tmp_attr0, d_aggr0__tmp_attr1, d_aggr0__tmp_attr2, d_aggr0__tmp_attr4, d_aggr0__tmp_attr9, d_aggr_rw__rw0, d_aggr_rw__rw1, d_aggr_rw__rw2, d_aggr_rw__rw3, d_aggr_rw__rw4, d_aggr_rw__rw5, d_KEY_57c908d0d780lineitem__l_linestatus, d_KEY_57c908d0d780lineitem__l_returnflag);
hipMemcpy(MAT57c908cee9c0lineitem__l_returnflag, d_MAT57c908cee9c0lineitem__l_returnflag, sizeof(DBCharType) * COUNT57c908cee9c0, hipMemcpyDeviceToHost);
hipMemcpy(MAT57c908cee9c0lineitem__l_linestatus, d_MAT57c908cee9c0lineitem__l_linestatus, sizeof(DBCharType) * COUNT57c908cee9c0, hipMemcpyDeviceToHost);
hipMemcpy(MAT57c908cee9c0aggr0__tmp_attr0, d_MAT57c908cee9c0aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT57c908cee9c0, hipMemcpyDeviceToHost);
hipMemcpy(MAT57c908cee9c0aggr0__tmp_attr1, d_MAT57c908cee9c0aggr0__tmp_attr1, sizeof(DBDecimalType) * COUNT57c908cee9c0, hipMemcpyDeviceToHost);
hipMemcpy(MAT57c908cee9c0aggr0__tmp_attr2, d_MAT57c908cee9c0aggr0__tmp_attr2, sizeof(DBDecimalType) * COUNT57c908cee9c0, hipMemcpyDeviceToHost);
hipMemcpy(MAT57c908cee9c0aggr0__tmp_attr4, d_MAT57c908cee9c0aggr0__tmp_attr4, sizeof(DBDecimalType) * COUNT57c908cee9c0, hipMemcpyDeviceToHost);
hipMemcpy(MAT57c908cee9c0aggr0__tmp_attr6, d_MAT57c908cee9c0aggr0__tmp_attr6, sizeof(DBDecimalType) * COUNT57c908cee9c0, hipMemcpyDeviceToHost);
hipMemcpy(MAT57c908cee9c0aggr0__tmp_attr7, d_MAT57c908cee9c0aggr0__tmp_attr7, sizeof(DBDecimalType) * COUNT57c908cee9c0, hipMemcpyDeviceToHost);
hipMemcpy(MAT57c908cee9c0aggr0__tmp_attr8, d_MAT57c908cee9c0aggr0__tmp_attr8, sizeof(DBDecimalType) * COUNT57c908cee9c0, hipMemcpyDeviceToHost);
hipMemcpy(MAT57c908cee9c0aggr0__tmp_attr9, d_MAT57c908cee9c0aggr0__tmp_attr9, sizeof(DBI64Type) * COUNT57c908cee9c0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT57c908cee9c0; i++) { std::cout << MAT57c908cee9c0lineitem__l_returnflag[i] << "\t";
std::cout << MAT57c908cee9c0lineitem__l_linestatus[i] << "\t";
std::cout << MAT57c908cee9c0aggr0__tmp_attr0[i] << "\t";
std::cout << MAT57c908cee9c0aggr0__tmp_attr1[i] << "\t";
std::cout << MAT57c908cee9c0aggr0__tmp_attr2[i] << "\t";
std::cout << MAT57c908cee9c0aggr0__tmp_attr4[i] << "\t";
std::cout << MAT57c908cee9c0aggr0__tmp_attr6[i] << "\t";
std::cout << MAT57c908cee9c0aggr0__tmp_attr7[i] << "\t";
std::cout << MAT57c908cee9c0aggr0__tmp_attr8[i] << "\t";
std::cout << MAT57c908cee9c0aggr0__tmp_attr9[i] << "\t";
std::cout << std::endl; }
hipFree(d_KEY_57c908d0d780lineitem__l_linestatus);
hipFree(d_KEY_57c908d0d780lineitem__l_returnflag);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr0__tmp_attr1);
hipFree(d_aggr0__tmp_attr2);
hipFree(d_aggr0__tmp_attr4);
hipFree(d_aggr0__tmp_attr9);
hipFree(d_aggr_rw__rw0);
hipFree(d_aggr_rw__rw1);
hipFree(d_aggr_rw__rw2);
hipFree(d_aggr_rw__rw3);
hipFree(d_aggr_rw__rw4);
hipFree(d_aggr_rw__rw5);
hipFree(d_COUNT57c908cee9c0);
hipFree(d_MAT57c908cee9c0aggr0__tmp_attr0);
hipFree(d_MAT57c908cee9c0aggr0__tmp_attr1);
hipFree(d_MAT57c908cee9c0aggr0__tmp_attr2);
hipFree(d_MAT57c908cee9c0aggr0__tmp_attr4);
hipFree(d_MAT57c908cee9c0aggr0__tmp_attr6);
hipFree(d_MAT57c908cee9c0aggr0__tmp_attr7);
hipFree(d_MAT57c908cee9c0aggr0__tmp_attr8);
hipFree(d_MAT57c908cee9c0aggr0__tmp_attr9);
hipFree(d_MAT57c908cee9c0lineitem__l_linestatus);
hipFree(d_MAT57c908cee9c0lineitem__l_returnflag);
hipFree(d_MAT_IDX57c908cee9c0);
free(MAT57c908cee9c0aggr0__tmp_attr0);
free(MAT57c908cee9c0aggr0__tmp_attr1);
free(MAT57c908cee9c0aggr0__tmp_attr2);
free(MAT57c908cee9c0aggr0__tmp_attr4);
free(MAT57c908cee9c0aggr0__tmp_attr6);
free(MAT57c908cee9c0aggr0__tmp_attr7);
free(MAT57c908cee9c0aggr0__tmp_attr8);
free(MAT57c908cee9c0aggr0__tmp_attr9);
free(MAT57c908cee9c0lineitem__l_linestatus);
free(MAT57c908cee9c0lineitem__l_returnflag);
}