#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5c784e87bbf0(uint64_t* COUNT5c784e88dc70, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8674, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::lt))) return;
//Materialize count
atomicAdd((int*)COUNT5c784e88dc70, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5c784e87bbf0(uint64_t* BUF_5c784e88dc70, uint64_t* BUF_IDX_5c784e88dc70, HASHTABLE_INSERT HT_5c784e88dc70, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8674, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::lt))) return;
uint64_t KEY_5c784e88dc70 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_5c784e88dc70 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_5c784e88dc70 = atomicAdd((int*)BUF_IDX_5c784e88dc70, 1);
HT_5c784e88dc70.insert(cuco::pair{KEY_5c784e88dc70, buf_idx_5c784e88dc70});
BUF_5c784e88dc70[buf_idx_5c784e88dc70 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5c784e89ab40(uint64_t* BUF_5c784e88dc70, uint64_t* COUNT5c784e88e120, HASHTABLE_PROBE HT_5c784e88dc70, DBI32Type* lineitem__l_orderkey, DBCharType* lineitem__l_returnflag, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];
if (!(evaluatePredicate(reg_lineitem__l_returnflag, "R", Predicate::eq))) return;
uint64_t KEY_5c784e88dc70 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5c784e88dc70 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5c784e88dc70.for_each(KEY_5c784e88dc70, [&] __device__ (auto const SLOT_5c784e88dc70) {

auto const [slot_first5c784e88dc70, slot_second5c784e88dc70] = SLOT_5c784e88dc70;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5c784e88e120, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5c784e89ab40(uint64_t* BUF_5c784e88dc70, uint64_t* BUF_5c784e88e120, uint64_t* BUF_IDX_5c784e88e120, HASHTABLE_PROBE HT_5c784e88dc70, HASHTABLE_INSERT HT_5c784e88e120, DBI32Type* lineitem__l_orderkey, DBCharType* lineitem__l_returnflag, size_t lineitem_size, DBI32Type* orders__o_custkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];
if (!(evaluatePredicate(reg_lineitem__l_returnflag, "R", Predicate::eq))) return;
uint64_t KEY_5c784e88dc70 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5c784e88dc70 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5c784e88dc70.for_each(KEY_5c784e88dc70, [&] __device__ (auto const SLOT_5c784e88dc70) {
auto const [slot_first5c784e88dc70, slot_second5c784e88dc70] = SLOT_5c784e88dc70;
if (!(true)) return;
uint64_t KEY_5c784e88e120 = 0;
auto reg_orders__o_custkey = orders__o_custkey[BUF_5c784e88dc70[slot_second5c784e88dc70 * 1 + 0]];

KEY_5c784e88e120 |= reg_orders__o_custkey;
// Insert hash table kernel;
auto buf_idx_5c784e88e120 = atomicAdd((int*)BUF_IDX_5c784e88e120, 1);
HT_5c784e88e120.insert(cuco::pair{KEY_5c784e88e120, buf_idx_5c784e88e120});
BUF_5c784e88e120[buf_idx_5c784e88e120 * 2 + 0] = BUF_5c784e88dc70[slot_second5c784e88dc70 * 1 + 0];
BUF_5c784e88e120[buf_idx_5c784e88e120 * 2 + 1] = tid;
});
}
__global__ void count_5c784e8a42d0(uint64_t* COUNT5c784e890710, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
//Materialize count
atomicAdd((int*)COUNT5c784e890710, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5c784e8a42d0(uint64_t* BUF_5c784e890710, uint64_t* BUF_IDX_5c784e890710, HASHTABLE_INSERT HT_5c784e890710, DBI32Type* nation__n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_5c784e890710 = 0;
auto reg_nation__n_nationkey = nation__n_nationkey[tid];

KEY_5c784e890710 |= reg_nation__n_nationkey;
// Insert hash table kernel;
auto buf_idx_5c784e890710 = atomicAdd((int*)BUF_IDX_5c784e890710, 1);
HT_5c784e890710.insert(cuco::pair{KEY_5c784e890710, buf_idx_5c784e890710});
BUF_5c784e890710[buf_idx_5c784e890710 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5c784e87b860(uint64_t* BUF_5c784e88e120, uint64_t* BUF_5c784e890710, HASHTABLE_INSERT HT_5c784e84b800, HASHTABLE_PROBE HT_5c784e88e120, HASHTABLE_PROBE HT_5c784e890710, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_5c784e88e120 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5c784e88e120 |= reg_customer__c_custkey;
//Probe Hash table
HT_5c784e88e120.for_each(KEY_5c784e88e120, [&] __device__ (auto const SLOT_5c784e88e120) {

auto const [slot_first5c784e88e120, slot_second5c784e88e120] = SLOT_5c784e88e120;
if (!(true)) return;
uint64_t KEY_5c784e890710 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_5c784e890710 |= reg_customer__c_nationkey;
//Probe Hash table
HT_5c784e890710.for_each(KEY_5c784e890710, [&] __device__ (auto const SLOT_5c784e890710) {

auto const [slot_first5c784e890710, slot_second5c784e890710] = SLOT_5c784e890710;
if (!(true)) return;
uint64_t KEY_5c784e84b800 = 0;

KEY_5c784e84b800 |= reg_customer__c_custkey;
//Create aggregation hash table
HT_5c784e84b800.insert(cuco::pair{KEY_5c784e84b800, 1});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5c784e87b860(uint64_t* BUF_5c784e88e120, uint64_t* BUF_5c784e890710, HASHTABLE_FIND HT_5c784e84b800, HASHTABLE_PROBE HT_5c784e88e120, HASHTABLE_PROBE HT_5c784e890710, DBI32Type* KEY_5c784e84b800customer__c_custkey, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr__c_acctbal, DBStringType* aggr__c_address, DBStringType* aggr__c_comment, DBStringType* aggr__c_name, DBStringType* aggr__c_phone, DBStringType* aggr__n_name, DBDecimalType* customer__c_acctbal, DBStringType* customer__c_address, DBStringType* customer__c_comment, DBI32Type* customer__c_custkey, DBStringType* customer__c_name, DBI32Type* customer__c_nationkey, DBStringType* customer__c_phone, size_t customer_size, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBStringType* nation__n_name) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_5c784e88e120 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5c784e88e120 |= reg_customer__c_custkey;
//Probe Hash table
HT_5c784e88e120.for_each(KEY_5c784e88e120, [&] __device__ (auto const SLOT_5c784e88e120) {
auto const [slot_first5c784e88e120, slot_second5c784e88e120] = SLOT_5c784e88e120;
if (!(true)) return;
uint64_t KEY_5c784e890710 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_5c784e890710 |= reg_customer__c_nationkey;
//Probe Hash table
HT_5c784e890710.for_each(KEY_5c784e890710, [&] __device__ (auto const SLOT_5c784e890710) {
auto const [slot_first5c784e890710, slot_second5c784e890710] = SLOT_5c784e890710;
if (!(true)) return;
uint64_t KEY_5c784e84b800 = 0;

KEY_5c784e84b800 |= reg_customer__c_custkey;
//Aggregate in hashtable
auto buf_idx_5c784e84b800 = HT_5c784e84b800.find(KEY_5c784e84b800)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[BUF_5c784e88e120[slot_second5c784e88e120 * 2 + 1]];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[BUF_5c784e88e120[slot_second5c784e88e120 * 2 + 1]];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5c784e84b800], reg_map0__tmp_attr1);
auto reg_customer__c_comment = customer__c_comment[tid];
aggregate_any(&aggr__c_comment[buf_idx_5c784e84b800], reg_customer__c_comment);
auto reg_customer__c_name = customer__c_name[tid];
aggregate_any(&aggr__c_name[buf_idx_5c784e84b800], reg_customer__c_name);
auto reg_customer__c_acctbal = customer__c_acctbal[tid];
aggregate_any(&aggr__c_acctbal[buf_idx_5c784e84b800], reg_customer__c_acctbal);
auto reg_customer__c_phone = customer__c_phone[tid];
aggregate_any(&aggr__c_phone[buf_idx_5c784e84b800], reg_customer__c_phone);
auto reg_nation__n_name = nation__n_name[BUF_5c784e890710[slot_second5c784e890710 * 1 + 0]];
aggregate_any(&aggr__n_name[buf_idx_5c784e84b800], reg_nation__n_name);
auto reg_customer__c_address = customer__c_address[tid];
aggregate_any(&aggr__c_address[buf_idx_5c784e84b800], reg_customer__c_address);
KEY_5c784e84b800customer__c_custkey[buf_idx_5c784e84b800] = reg_customer__c_custkey;
});
});
}
__global__ void count_5c784e8aa8e0(uint64_t* COUNT5c784e828c60, size_t COUNT5c784e84b800) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5c784e84b800) return;
//Materialize count
atomicAdd((int*)COUNT5c784e828c60, 1);
}
__global__ void main_5c784e8aa8e0(size_t COUNT5c784e84b800, DBDecimalType* MAT5c784e828c60aggr0__tmp_attr0, DBDecimalType* MAT5c784e828c60aggr__c_acctbal, DBStringType* MAT5c784e828c60aggr__c_address, DBStringType* MAT5c784e828c60aggr__c_comment, DBStringType* MAT5c784e828c60aggr__c_name, DBStringType* MAT5c784e828c60aggr__c_phone, DBStringType* MAT5c784e828c60aggr__n_name, DBI32Type* MAT5c784e828c60customer__c_custkey, uint64_t* MAT_IDX5c784e828c60, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr__c_acctbal, DBStringType* aggr__c_address, DBStringType* aggr__c_comment, DBStringType* aggr__c_name, DBStringType* aggr__c_phone, DBStringType* aggr__n_name, DBI32Type* customer__c_custkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5c784e84b800) return;
//Materialize buffers
auto mat_idx5c784e828c60 = atomicAdd((int*)MAT_IDX5c784e828c60, 1);
auto reg_customer__c_custkey = customer__c_custkey[tid];
MAT5c784e828c60customer__c_custkey[mat_idx5c784e828c60] = reg_customer__c_custkey;
auto reg_aggr__c_name = aggr__c_name[tid];
MAT5c784e828c60aggr__c_name[mat_idx5c784e828c60] = reg_aggr__c_name;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5c784e828c60aggr0__tmp_attr0[mat_idx5c784e828c60] = reg_aggr0__tmp_attr0;
auto reg_aggr__c_acctbal = aggr__c_acctbal[tid];
MAT5c784e828c60aggr__c_acctbal[mat_idx5c784e828c60] = reg_aggr__c_acctbal;
auto reg_aggr__n_name = aggr__n_name[tid];
MAT5c784e828c60aggr__n_name[mat_idx5c784e828c60] = reg_aggr__n_name;
auto reg_aggr__c_address = aggr__c_address[tid];
MAT5c784e828c60aggr__c_address[mat_idx5c784e828c60] = reg_aggr__c_address;
auto reg_aggr__c_phone = aggr__c_phone[tid];
MAT5c784e828c60aggr__c_phone[mat_idx5c784e828c60] = reg_aggr__c_phone;
auto reg_aggr__c_comment = aggr__c_comment[tid];
MAT5c784e828c60aggr__c_comment[mat_idx5c784e828c60] = reg_aggr__c_comment;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT5c784e88dc70;
hipMalloc(&d_COUNT5c784e88dc70, sizeof(uint64_t));
hipMemset(d_COUNT5c784e88dc70, 0, sizeof(uint64_t));
count_5c784e87bbf0<<<std::ceil((float)orders_size/32.), 32>>>(d_COUNT5c784e88dc70, d_orders__o_orderdate, orders_size);
uint64_t COUNT5c784e88dc70;
hipMemcpy(&COUNT5c784e88dc70, d_COUNT5c784e88dc70, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5c784e88dc70;
hipMalloc(&d_BUF_IDX_5c784e88dc70, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5c784e88dc70, 0, sizeof(uint64_t));
uint64_t* d_BUF_5c784e88dc70;
hipMalloc(&d_BUF_5c784e88dc70, sizeof(uint64_t) * COUNT5c784e88dc70 * 1);
auto d_HT_5c784e88dc70 = cuco::experimental::static_multimap{ (int)COUNT5c784e88dc70*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5c784e87bbf0<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5c784e88dc70, d_BUF_IDX_5c784e88dc70, d_HT_5c784e88dc70.ref(cuco::insert), d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
//Materialize count
uint64_t* d_COUNT5c784e88e120;
hipMalloc(&d_COUNT5c784e88e120, sizeof(uint64_t));
hipMemset(d_COUNT5c784e88e120, 0, sizeof(uint64_t));
count_5c784e89ab40<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5c784e88dc70, d_COUNT5c784e88e120, d_HT_5c784e88dc70.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_returnflag, lineitem_size);
uint64_t COUNT5c784e88e120;
hipMemcpy(&COUNT5c784e88e120, d_COUNT5c784e88e120, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5c784e88e120;
hipMalloc(&d_BUF_IDX_5c784e88e120, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5c784e88e120, 0, sizeof(uint64_t));
uint64_t* d_BUF_5c784e88e120;
hipMalloc(&d_BUF_5c784e88e120, sizeof(uint64_t) * COUNT5c784e88e120 * 2);
auto d_HT_5c784e88e120 = cuco::experimental::static_multimap{ (int)COUNT5c784e88e120*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5c784e89ab40<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5c784e88dc70, d_BUF_5c784e88e120, d_BUF_IDX_5c784e88e120, d_HT_5c784e88dc70.ref(cuco::for_each), d_HT_5c784e88e120.ref(cuco::insert), d_lineitem__l_orderkey, d_lineitem__l_returnflag, lineitem_size, d_orders__o_custkey);
//Materialize count
uint64_t* d_COUNT5c784e890710;
hipMalloc(&d_COUNT5c784e890710, sizeof(uint64_t));
hipMemset(d_COUNT5c784e890710, 0, sizeof(uint64_t));
count_5c784e8a42d0<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT5c784e890710, nation_size);
uint64_t COUNT5c784e890710;
hipMemcpy(&COUNT5c784e890710, d_COUNT5c784e890710, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_5c784e890710;
hipMalloc(&d_BUF_IDX_5c784e890710, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5c784e890710, 0, sizeof(uint64_t));
uint64_t* d_BUF_5c784e890710;
hipMalloc(&d_BUF_5c784e890710, sizeof(uint64_t) * COUNT5c784e890710 * 1);
auto d_HT_5c784e890710 = cuco::experimental::static_multimap{ (int)COUNT5c784e890710*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5c784e8a42d0<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_5c784e890710, d_BUF_IDX_5c784e890710, d_HT_5c784e890710.ref(cuco::insert), d_nation__n_nationkey, nation_size);
//Create aggregation hash table
auto d_HT_5c784e84b800 = cuco::static_map{ (int)45145*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5c784e87b860<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5c784e88e120, d_BUF_5c784e890710, d_HT_5c784e84b800.ref(cuco::insert), d_HT_5c784e88e120.ref(cuco::for_each), d_HT_5c784e890710.ref(cuco::for_each), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
size_t COUNT5c784e84b800 = d_HT_5c784e84b800.size();
thrust::device_vector<int64_t> keys_5c784e84b800(COUNT5c784e84b800), vals_5c784e84b800(COUNT5c784e84b800);
d_HT_5c784e84b800.retrieve_all(keys_5c784e84b800.begin(), vals_5c784e84b800.begin());
d_HT_5c784e84b800.clear();
int64_t* raw_keys5c784e84b800 = thrust::raw_pointer_cast(keys_5c784e84b800.data());
insertKeys<<<std::ceil((float)COUNT5c784e84b800/32.), 32>>>(raw_keys5c784e84b800, d_HT_5c784e84b800.ref(cuco::insert), COUNT5c784e84b800);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5c784e84b800);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5c784e84b800);
DBStringType* d_aggr__c_comment;
hipMalloc(&d_aggr__c_comment, sizeof(DBStringType) * COUNT5c784e84b800);
hipMemset(d_aggr__c_comment, 0, sizeof(DBStringType) * COUNT5c784e84b800);
DBStringType* d_aggr__c_name;
hipMalloc(&d_aggr__c_name, sizeof(DBStringType) * COUNT5c784e84b800);
hipMemset(d_aggr__c_name, 0, sizeof(DBStringType) * COUNT5c784e84b800);
DBDecimalType* d_aggr__c_acctbal;
hipMalloc(&d_aggr__c_acctbal, sizeof(DBDecimalType) * COUNT5c784e84b800);
hipMemset(d_aggr__c_acctbal, 0, sizeof(DBDecimalType) * COUNT5c784e84b800);
DBStringType* d_aggr__c_phone;
hipMalloc(&d_aggr__c_phone, sizeof(DBStringType) * COUNT5c784e84b800);
hipMemset(d_aggr__c_phone, 0, sizeof(DBStringType) * COUNT5c784e84b800);
DBStringType* d_aggr__n_name;
hipMalloc(&d_aggr__n_name, sizeof(DBStringType) * COUNT5c784e84b800);
hipMemset(d_aggr__n_name, 0, sizeof(DBStringType) * COUNT5c784e84b800);
DBStringType* d_aggr__c_address;
hipMalloc(&d_aggr__c_address, sizeof(DBStringType) * COUNT5c784e84b800);
hipMemset(d_aggr__c_address, 0, sizeof(DBStringType) * COUNT5c784e84b800);
DBI32Type* d_KEY_5c784e84b800customer__c_custkey;
hipMalloc(&d_KEY_5c784e84b800customer__c_custkey, sizeof(DBI32Type) * COUNT5c784e84b800);
hipMemset(d_KEY_5c784e84b800customer__c_custkey, 0, sizeof(DBI32Type) * COUNT5c784e84b800);
main_5c784e87b860<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5c784e88e120, d_BUF_5c784e890710, d_HT_5c784e84b800.ref(cuco::find), d_HT_5c784e88e120.ref(cuco::for_each), d_HT_5c784e890710.ref(cuco::for_each), d_KEY_5c784e84b800customer__c_custkey, d_aggr0__tmp_attr0, d_aggr__c_acctbal, d_aggr__c_address, d_aggr__c_comment, d_aggr__c_name, d_aggr__c_phone, d_aggr__n_name, d_customer__c_acctbal, d_customer__c_address, d_customer__c_comment, d_customer__c_custkey, d_customer__c_name, d_customer__c_nationkey, d_customer__c_phone, customer_size, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_nation__n_name);
//Materialize count
uint64_t* d_COUNT5c784e828c60;
hipMalloc(&d_COUNT5c784e828c60, sizeof(uint64_t));
hipMemset(d_COUNT5c784e828c60, 0, sizeof(uint64_t));
count_5c784e8aa8e0<<<std::ceil((float)COUNT5c784e84b800/32.), 32>>>(d_COUNT5c784e828c60, COUNT5c784e84b800);
uint64_t COUNT5c784e828c60;
hipMemcpy(&COUNT5c784e828c60, d_COUNT5c784e828c60, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX5c784e828c60;
hipMalloc(&d_MAT_IDX5c784e828c60, sizeof(uint64_t));
hipMemset(d_MAT_IDX5c784e828c60, 0, sizeof(uint64_t));
auto MAT5c784e828c60customer__c_custkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5c784e828c60);
DBI32Type* d_MAT5c784e828c60customer__c_custkey;
hipMalloc(&d_MAT5c784e828c60customer__c_custkey, sizeof(DBI32Type) * COUNT5c784e828c60);
auto MAT5c784e828c60aggr__c_name = (DBStringType*)malloc(sizeof(DBStringType) * COUNT5c784e828c60);
DBStringType* d_MAT5c784e828c60aggr__c_name;
hipMalloc(&d_MAT5c784e828c60aggr__c_name, sizeof(DBStringType) * COUNT5c784e828c60);
auto MAT5c784e828c60aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5c784e828c60);
DBDecimalType* d_MAT5c784e828c60aggr0__tmp_attr0;
hipMalloc(&d_MAT5c784e828c60aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5c784e828c60);
auto MAT5c784e828c60aggr__c_acctbal = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5c784e828c60);
DBDecimalType* d_MAT5c784e828c60aggr__c_acctbal;
hipMalloc(&d_MAT5c784e828c60aggr__c_acctbal, sizeof(DBDecimalType) * COUNT5c784e828c60);
auto MAT5c784e828c60aggr__n_name = (DBStringType*)malloc(sizeof(DBStringType) * COUNT5c784e828c60);
DBStringType* d_MAT5c784e828c60aggr__n_name;
hipMalloc(&d_MAT5c784e828c60aggr__n_name, sizeof(DBStringType) * COUNT5c784e828c60);
auto MAT5c784e828c60aggr__c_address = (DBStringType*)malloc(sizeof(DBStringType) * COUNT5c784e828c60);
DBStringType* d_MAT5c784e828c60aggr__c_address;
hipMalloc(&d_MAT5c784e828c60aggr__c_address, sizeof(DBStringType) * COUNT5c784e828c60);
auto MAT5c784e828c60aggr__c_phone = (DBStringType*)malloc(sizeof(DBStringType) * COUNT5c784e828c60);
DBStringType* d_MAT5c784e828c60aggr__c_phone;
hipMalloc(&d_MAT5c784e828c60aggr__c_phone, sizeof(DBStringType) * COUNT5c784e828c60);
auto MAT5c784e828c60aggr__c_comment = (DBStringType*)malloc(sizeof(DBStringType) * COUNT5c784e828c60);
DBStringType* d_MAT5c784e828c60aggr__c_comment;
hipMalloc(&d_MAT5c784e828c60aggr__c_comment, sizeof(DBStringType) * COUNT5c784e828c60);
main_5c784e8aa8e0<<<std::ceil((float)COUNT5c784e84b800/32.), 32>>>(COUNT5c784e84b800, d_MAT5c784e828c60aggr0__tmp_attr0, d_MAT5c784e828c60aggr__c_acctbal, d_MAT5c784e828c60aggr__c_address, d_MAT5c784e828c60aggr__c_comment, d_MAT5c784e828c60aggr__c_name, d_MAT5c784e828c60aggr__c_phone, d_MAT5c784e828c60aggr__n_name, d_MAT5c784e828c60customer__c_custkey, d_MAT_IDX5c784e828c60, d_aggr0__tmp_attr0, d_aggr__c_acctbal, d_aggr__c_address, d_aggr__c_comment, d_aggr__c_name, d_aggr__c_phone, d_aggr__n_name, d_KEY_5c784e84b800customer__c_custkey);
hipMemcpy(MAT5c784e828c60customer__c_custkey, d_MAT5c784e828c60customer__c_custkey, sizeof(DBI32Type) * COUNT5c784e828c60, hipMemcpyDeviceToHost);
hipMemcpy(MAT5c784e828c60aggr__c_name, d_MAT5c784e828c60aggr__c_name, sizeof(DBStringType) * COUNT5c784e828c60, hipMemcpyDeviceToHost);
hipMemcpy(MAT5c784e828c60aggr0__tmp_attr0, d_MAT5c784e828c60aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5c784e828c60, hipMemcpyDeviceToHost);
hipMemcpy(MAT5c784e828c60aggr__c_acctbal, d_MAT5c784e828c60aggr__c_acctbal, sizeof(DBDecimalType) * COUNT5c784e828c60, hipMemcpyDeviceToHost);
hipMemcpy(MAT5c784e828c60aggr__n_name, d_MAT5c784e828c60aggr__n_name, sizeof(DBStringType) * COUNT5c784e828c60, hipMemcpyDeviceToHost);
hipMemcpy(MAT5c784e828c60aggr__c_address, d_MAT5c784e828c60aggr__c_address, sizeof(DBStringType) * COUNT5c784e828c60, hipMemcpyDeviceToHost);
hipMemcpy(MAT5c784e828c60aggr__c_phone, d_MAT5c784e828c60aggr__c_phone, sizeof(DBStringType) * COUNT5c784e828c60, hipMemcpyDeviceToHost);
hipMemcpy(MAT5c784e828c60aggr__c_comment, d_MAT5c784e828c60aggr__c_comment, sizeof(DBStringType) * COUNT5c784e828c60, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5c784e828c60; i++) { std::cout << MAT5c784e828c60customer__c_custkey[i] << "\t";
std::cout << MAT5c784e828c60aggr__c_name[i] << "\t";
std::cout << MAT5c784e828c60aggr0__tmp_attr0[i] << "\t";
std::cout << MAT5c784e828c60aggr__c_acctbal[i] << "\t";
std::cout << MAT5c784e828c60aggr__n_name[i] << "\t";
std::cout << MAT5c784e828c60aggr__c_address[i] << "\t";
std::cout << MAT5c784e828c60aggr__c_phone[i] << "\t";
std::cout << MAT5c784e828c60aggr__c_comment[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_5c784e88dc70);
hipFree(d_BUF_IDX_5c784e88dc70);
hipFree(d_COUNT5c784e88dc70);
hipFree(d_BUF_5c784e88e120);
hipFree(d_BUF_IDX_5c784e88e120);
hipFree(d_COUNT5c784e88e120);
hipFree(d_BUF_5c784e890710);
hipFree(d_BUF_IDX_5c784e890710);
hipFree(d_COUNT5c784e890710);
hipFree(d_KEY_5c784e84b800customer__c_custkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_aggr__c_acctbal);
hipFree(d_aggr__c_address);
hipFree(d_aggr__c_comment);
hipFree(d_aggr__c_name);
hipFree(d_aggr__c_phone);
hipFree(d_aggr__n_name);
hipFree(d_COUNT5c784e828c60);
hipFree(d_MAT5c784e828c60aggr0__tmp_attr0);
hipFree(d_MAT5c784e828c60aggr__c_acctbal);
hipFree(d_MAT5c784e828c60aggr__c_address);
hipFree(d_MAT5c784e828c60aggr__c_comment);
hipFree(d_MAT5c784e828c60aggr__c_name);
hipFree(d_MAT5c784e828c60aggr__c_phone);
hipFree(d_MAT5c784e828c60aggr__n_name);
hipFree(d_MAT5c784e828c60customer__c_custkey);
hipFree(d_MAT_IDX5c784e828c60);
free(MAT5c784e828c60aggr0__tmp_attr0);
free(MAT5c784e828c60aggr__c_acctbal);
free(MAT5c784e828c60aggr__c_address);
free(MAT5c784e828c60aggr__c_comment);
free(MAT5c784e828c60aggr__c_name);
free(MAT5c784e828c60aggr__c_phone);
free(MAT5c784e828c60aggr__n_name);
free(MAT5c784e828c60customer__c_custkey);
}