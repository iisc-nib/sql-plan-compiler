#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5abfc2a8e790(uint64_t* COUNT5abfc2b012c0, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8674, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::lt))) return;
//Materialize count
atomicAdd((int*)COUNT5abfc2b012c0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5abfc2a8e790(uint64_t* BUF_5abfc2b012c0, uint64_t* BUF_IDX_5abfc2b012c0, HASHTABLE_INSERT HT_5abfc2b012c0, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 8674, Predicate::gte) && evaluatePredicate(reg_orders__o_orderdate, 8766, Predicate::lt))) return;
uint64_t KEY_5abfc2b012c0 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_5abfc2b012c0 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_5abfc2b012c0 = atomicAdd((int*)BUF_IDX_5abfc2b012c0, 1);
HT_5abfc2b012c0.insert(cuco::pair{KEY_5abfc2b012c0, buf_idx_5abfc2b012c0});
BUF_5abfc2b012c0[buf_idx_5abfc2b012c0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5abfc2b0d980(uint64_t* BUF_5abfc2b012c0, uint64_t* COUNT5abfc2b014a0, HASHTABLE_PROBE HT_5abfc2b012c0, DBI32Type* lineitem__l_orderkey, DBCharType* lineitem__l_returnflag, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];
if (!(evaluatePredicate(reg_lineitem__l_returnflag, 'R', Predicate::eq))) return;
uint64_t KEY_5abfc2b012c0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5abfc2b012c0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5abfc2b012c0.for_each(KEY_5abfc2b012c0, [&] __device__ (auto const SLOT_5abfc2b012c0) {

auto const [slot_first5abfc2b012c0, slot_second5abfc2b012c0] = SLOT_5abfc2b012c0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5abfc2b014a0, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5abfc2b0d980(uint64_t* BUF_5abfc2b012c0, uint64_t* BUF_5abfc2b014a0, uint64_t* BUF_IDX_5abfc2b014a0, HASHTABLE_PROBE HT_5abfc2b012c0, HASHTABLE_INSERT HT_5abfc2b014a0, DBI32Type* lineitem__l_orderkey, DBCharType* lineitem__l_returnflag, size_t lineitem_size, DBI32Type* orders__o_custkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_returnflag = lineitem__l_returnflag[tid];
if (!(evaluatePredicate(reg_lineitem__l_returnflag, 'R', Predicate::eq))) return;
uint64_t KEY_5abfc2b012c0 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5abfc2b012c0 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5abfc2b012c0.for_each(KEY_5abfc2b012c0, [&] __device__ (auto const SLOT_5abfc2b012c0) {
auto const [slot_first5abfc2b012c0, slot_second5abfc2b012c0] = SLOT_5abfc2b012c0;
if (!(true)) return;
uint64_t KEY_5abfc2b014a0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[BUF_5abfc2b012c0[slot_second5abfc2b012c0 * 1 + 0]];

KEY_5abfc2b014a0 |= reg_orders__o_custkey;
// Insert hash table kernel;
auto buf_idx_5abfc2b014a0 = atomicAdd((int*)BUF_IDX_5abfc2b014a0, 1);
HT_5abfc2b014a0.insert(cuco::pair{KEY_5abfc2b014a0, buf_idx_5abfc2b014a0});
BUF_5abfc2b014a0[buf_idx_5abfc2b014a0 * 2 + 0] = BUF_5abfc2b012c0[slot_second5abfc2b012c0 * 1 + 0];
BUF_5abfc2b014a0[buf_idx_5abfc2b014a0 * 2 + 1] = tid;
});
}
__global__ void count_5abfc2b16930(uint64_t* COUNT5abfc2b03040, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
//Materialize count
atomicAdd((int*)COUNT5abfc2b03040, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5abfc2b16930(uint64_t* BUF_5abfc2b03040, uint64_t* BUF_IDX_5abfc2b03040, HASHTABLE_INSERT HT_5abfc2b03040, DBI32Type* nation__n_nationkey, size_t nation_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= nation_size) return;
uint64_t KEY_5abfc2b03040 = 0;
auto reg_nation__n_nationkey = nation__n_nationkey[tid];

KEY_5abfc2b03040 |= reg_nation__n_nationkey;
// Insert hash table kernel;
auto buf_idx_5abfc2b03040 = atomicAdd((int*)BUF_IDX_5abfc2b03040, 1);
HT_5abfc2b03040.insert(cuco::pair{KEY_5abfc2b03040, buf_idx_5abfc2b03040});
BUF_5abfc2b03040[buf_idx_5abfc2b03040 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5abfc2a0b910(uint64_t* BUF_5abfc2b014a0, uint64_t* BUF_5abfc2b03040, HASHTABLE_INSERT HT_5abfc2abe530, HASHTABLE_PROBE HT_5abfc2b014a0, HASHTABLE_PROBE HT_5abfc2b03040, DBI32Type* customer__c_custkey, DBI32Type* customer__c_nationkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_5abfc2b014a0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5abfc2b014a0 |= reg_customer__c_custkey;
//Probe Hash table
HT_5abfc2b014a0.for_each(KEY_5abfc2b014a0, [&] __device__ (auto const SLOT_5abfc2b014a0) {

auto const [slot_first5abfc2b014a0, slot_second5abfc2b014a0] = SLOT_5abfc2b014a0;
if (!(true)) return;
uint64_t KEY_5abfc2b03040 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_5abfc2b03040 |= reg_customer__c_nationkey;
//Probe Hash table
HT_5abfc2b03040.for_each(KEY_5abfc2b03040, [&] __device__ (auto const SLOT_5abfc2b03040) {

auto const [slot_first5abfc2b03040, slot_second5abfc2b03040] = SLOT_5abfc2b03040;
if (!(true)) return;
uint64_t KEY_5abfc2abe530 = 0;

KEY_5abfc2abe530 |= reg_customer__c_custkey;
//Create aggregation hash table
HT_5abfc2abe530.insert(cuco::pair{KEY_5abfc2abe530, 1});
});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5abfc2a0b910(uint64_t* BUF_5abfc2b014a0, uint64_t* BUF_5abfc2b03040, HASHTABLE_FIND HT_5abfc2abe530, HASHTABLE_PROBE HT_5abfc2b014a0, HASHTABLE_PROBE HT_5abfc2b03040, DBI32Type* KEY_5abfc2abe530customer__c_custkey, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr__c_acctbal, DBStringType* aggr__c_address, DBStringType* aggr__c_comment, DBStringType* aggr__c_name, DBStringType* aggr__c_phone, DBStringType* aggr__n_name, DBDecimalType* customer__c_acctbal, DBStringType* customer__c_address, DBStringType* customer__c_comment, DBI32Type* customer__c_custkey, DBStringType* customer__c_name, DBI32Type* customer__c_nationkey, DBStringType* customer__c_phone, size_t customer_size, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBStringType* nation__n_name) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_5abfc2b014a0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5abfc2b014a0 |= reg_customer__c_custkey;
//Probe Hash table
HT_5abfc2b014a0.for_each(KEY_5abfc2b014a0, [&] __device__ (auto const SLOT_5abfc2b014a0) {
auto const [slot_first5abfc2b014a0, slot_second5abfc2b014a0] = SLOT_5abfc2b014a0;
if (!(true)) return;
uint64_t KEY_5abfc2b03040 = 0;
auto reg_customer__c_nationkey = customer__c_nationkey[tid];

KEY_5abfc2b03040 |= reg_customer__c_nationkey;
//Probe Hash table
HT_5abfc2b03040.for_each(KEY_5abfc2b03040, [&] __device__ (auto const SLOT_5abfc2b03040) {
auto const [slot_first5abfc2b03040, slot_second5abfc2b03040] = SLOT_5abfc2b03040;
if (!(true)) return;
uint64_t KEY_5abfc2abe530 = 0;

KEY_5abfc2abe530 |= reg_customer__c_custkey;
//Aggregate in hashtable
auto buf_idx_5abfc2abe530 = HT_5abfc2abe530.find(KEY_5abfc2abe530)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[BUF_5abfc2b014a0[slot_second5abfc2b014a0 * 2 + 1]];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[BUF_5abfc2b014a0[slot_second5abfc2b014a0 * 2 + 1]];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5abfc2abe530], reg_map0__tmp_attr1);
auto reg_customer__c_comment = customer__c_comment[tid];
aggregate_any(&aggr__c_comment[buf_idx_5abfc2abe530], reg_customer__c_comment);
auto reg_customer__c_name = customer__c_name[tid];
aggregate_any(&aggr__c_name[buf_idx_5abfc2abe530], reg_customer__c_name);
auto reg_customer__c_acctbal = customer__c_acctbal[tid];
aggregate_any(&aggr__c_acctbal[buf_idx_5abfc2abe530], reg_customer__c_acctbal);
auto reg_customer__c_phone = customer__c_phone[tid];
aggregate_any(&aggr__c_phone[buf_idx_5abfc2abe530], reg_customer__c_phone);
auto reg_nation__n_name = nation__n_name[BUF_5abfc2b03040[slot_second5abfc2b03040 * 1 + 0]];
aggregate_any(&aggr__n_name[buf_idx_5abfc2abe530], reg_nation__n_name);
auto reg_customer__c_address = customer__c_address[tid];
aggregate_any(&aggr__c_address[buf_idx_5abfc2abe530], reg_customer__c_address);
KEY_5abfc2abe530customer__c_custkey[buf_idx_5abfc2abe530] = reg_customer__c_custkey;
});
});
}
__global__ void count_5abfc2b1d700(size_t COUNT5abfc2abe530, uint64_t* COUNT5abfc2ad1890) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5abfc2abe530) return;
//Materialize count
atomicAdd((int*)COUNT5abfc2ad1890, 1);
}
__global__ void main_5abfc2b1d700(size_t COUNT5abfc2abe530, DBDecimalType* MAT5abfc2ad1890aggr0__tmp_attr0, DBDecimalType* MAT5abfc2ad1890aggr__c_acctbal, DBStringType* MAT5abfc2ad1890aggr__c_address, DBStringType* MAT5abfc2ad1890aggr__c_comment, DBStringType* MAT5abfc2ad1890aggr__c_name, DBStringType* MAT5abfc2ad1890aggr__c_phone, DBStringType* MAT5abfc2ad1890aggr__n_name, DBI32Type* MAT5abfc2ad1890customer__c_custkey, uint64_t* MAT_IDX5abfc2ad1890, DBDecimalType* aggr0__tmp_attr0, DBDecimalType* aggr__c_acctbal, DBStringType* aggr__c_address, DBStringType* aggr__c_comment, DBStringType* aggr__c_name, DBStringType* aggr__c_phone, DBStringType* aggr__n_name, DBI32Type* customer__c_custkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5abfc2abe530) return;
//Materialize buffers
auto mat_idx5abfc2ad1890 = atomicAdd((int*)MAT_IDX5abfc2ad1890, 1);
auto reg_customer__c_custkey = customer__c_custkey[tid];
MAT5abfc2ad1890customer__c_custkey[mat_idx5abfc2ad1890] = reg_customer__c_custkey;
auto reg_aggr__c_name = aggr__c_name[tid];
MAT5abfc2ad1890aggr__c_name[mat_idx5abfc2ad1890] = reg_aggr__c_name;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5abfc2ad1890aggr0__tmp_attr0[mat_idx5abfc2ad1890] = reg_aggr0__tmp_attr0;
auto reg_aggr__c_acctbal = aggr__c_acctbal[tid];
MAT5abfc2ad1890aggr__c_acctbal[mat_idx5abfc2ad1890] = reg_aggr__c_acctbal;
auto reg_aggr__n_name = aggr__n_name[tid];
MAT5abfc2ad1890aggr__n_name[mat_idx5abfc2ad1890] = reg_aggr__n_name;
auto reg_aggr__c_address = aggr__c_address[tid];
MAT5abfc2ad1890aggr__c_address[mat_idx5abfc2ad1890] = reg_aggr__c_address;
auto reg_aggr__c_phone = aggr__c_phone[tid];
MAT5abfc2ad1890aggr__c_phone[mat_idx5abfc2ad1890] = reg_aggr__c_phone;
auto reg_aggr__c_comment = aggr__c_comment[tid];
MAT5abfc2ad1890aggr__c_comment[mat_idx5abfc2ad1890] = reg_aggr__c_comment;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT5abfc2b012c0;
hipMalloc(&d_COUNT5abfc2b012c0, sizeof(uint64_t));
hipMemset(d_COUNT5abfc2b012c0, 0, sizeof(uint64_t));
count_5abfc2a8e790<<<std::ceil((float)orders_size/32.), 32>>>(d_COUNT5abfc2b012c0, d_orders__o_orderdate, orders_size);
uint64_t COUNT5abfc2b012c0;
hipMemcpy(&COUNT5abfc2b012c0, d_COUNT5abfc2b012c0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5abfc2b012c0);
// Insert hash table control;
uint64_t* d_BUF_IDX_5abfc2b012c0;
hipMalloc(&d_BUF_IDX_5abfc2b012c0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5abfc2b012c0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5abfc2b012c0;
hipMalloc(&d_BUF_5abfc2b012c0, sizeof(uint64_t) * COUNT5abfc2b012c0 * 1);
auto d_HT_5abfc2b012c0 = cuco::experimental::static_multimap{ (int)COUNT5abfc2b012c0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5abfc2a8e790<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5abfc2b012c0, d_BUF_IDX_5abfc2b012c0, d_HT_5abfc2b012c0.ref(cuco::insert), d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
hipFree(d_BUF_IDX_5abfc2b012c0);
//Materialize count
uint64_t* d_COUNT5abfc2b014a0;
hipMalloc(&d_COUNT5abfc2b014a0, sizeof(uint64_t));
hipMemset(d_COUNT5abfc2b014a0, 0, sizeof(uint64_t));
count_5abfc2b0d980<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5abfc2b012c0, d_COUNT5abfc2b014a0, d_HT_5abfc2b012c0.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_returnflag, lineitem_size);
uint64_t COUNT5abfc2b014a0;
hipMemcpy(&COUNT5abfc2b014a0, d_COUNT5abfc2b014a0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5abfc2b014a0);
// Insert hash table control;
uint64_t* d_BUF_IDX_5abfc2b014a0;
hipMalloc(&d_BUF_IDX_5abfc2b014a0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5abfc2b014a0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5abfc2b014a0;
hipMalloc(&d_BUF_5abfc2b014a0, sizeof(uint64_t) * COUNT5abfc2b014a0 * 2);
auto d_HT_5abfc2b014a0 = cuco::experimental::static_multimap{ (int)COUNT5abfc2b014a0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5abfc2b0d980<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5abfc2b012c0, d_BUF_5abfc2b014a0, d_BUF_IDX_5abfc2b014a0, d_HT_5abfc2b012c0.ref(cuco::for_each), d_HT_5abfc2b014a0.ref(cuco::insert), d_lineitem__l_orderkey, d_lineitem__l_returnflag, lineitem_size, d_orders__o_custkey);
hipFree(d_BUF_IDX_5abfc2b014a0);
//Materialize count
uint64_t* d_COUNT5abfc2b03040;
hipMalloc(&d_COUNT5abfc2b03040, sizeof(uint64_t));
hipMemset(d_COUNT5abfc2b03040, 0, sizeof(uint64_t));
count_5abfc2b16930<<<std::ceil((float)nation_size/32.), 32>>>(d_COUNT5abfc2b03040, nation_size);
uint64_t COUNT5abfc2b03040;
hipMemcpy(&COUNT5abfc2b03040, d_COUNT5abfc2b03040, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5abfc2b03040);
// Insert hash table control;
uint64_t* d_BUF_IDX_5abfc2b03040;
hipMalloc(&d_BUF_IDX_5abfc2b03040, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5abfc2b03040, 0, sizeof(uint64_t));
uint64_t* d_BUF_5abfc2b03040;
hipMalloc(&d_BUF_5abfc2b03040, sizeof(uint64_t) * COUNT5abfc2b03040 * 1);
auto d_HT_5abfc2b03040 = cuco::experimental::static_multimap{ (int)COUNT5abfc2b03040*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5abfc2b16930<<<std::ceil((float)nation_size/32.), 32>>>(d_BUF_5abfc2b03040, d_BUF_IDX_5abfc2b03040, d_HT_5abfc2b03040.ref(cuco::insert), d_nation__n_nationkey, nation_size);
hipFree(d_BUF_IDX_5abfc2b03040);
//Create aggregation hash table
auto d_HT_5abfc2abe530 = cuco::static_map{ (int)45145*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5abfc2a0b910<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5abfc2b014a0, d_BUF_5abfc2b03040, d_HT_5abfc2abe530.ref(cuco::insert), d_HT_5abfc2b014a0.ref(cuco::for_each), d_HT_5abfc2b03040.ref(cuco::for_each), d_customer__c_custkey, d_customer__c_nationkey, customer_size);
size_t COUNT5abfc2abe530 = d_HT_5abfc2abe530.size();
thrust::device_vector<int64_t> keys_5abfc2abe530(COUNT5abfc2abe530), vals_5abfc2abe530(COUNT5abfc2abe530);
d_HT_5abfc2abe530.retrieve_all(keys_5abfc2abe530.begin(), vals_5abfc2abe530.begin());
d_HT_5abfc2abe530.clear();
int64_t* raw_keys5abfc2abe530 = thrust::raw_pointer_cast(keys_5abfc2abe530.data());
insertKeys<<<std::ceil((float)COUNT5abfc2abe530/32.), 32>>>(raw_keys5abfc2abe530, d_HT_5abfc2abe530.ref(cuco::insert), COUNT5abfc2abe530);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5abfc2abe530);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5abfc2abe530);
DBStringType* d_aggr__c_comment;
hipMalloc(&d_aggr__c_comment, sizeof(DBStringType) * COUNT5abfc2abe530);
hipMemset(d_aggr__c_comment, 0, sizeof(DBStringType) * COUNT5abfc2abe530);
DBStringType* d_aggr__c_name;
hipMalloc(&d_aggr__c_name, sizeof(DBStringType) * COUNT5abfc2abe530);
hipMemset(d_aggr__c_name, 0, sizeof(DBStringType) * COUNT5abfc2abe530);
DBDecimalType* d_aggr__c_acctbal;
hipMalloc(&d_aggr__c_acctbal, sizeof(DBDecimalType) * COUNT5abfc2abe530);
hipMemset(d_aggr__c_acctbal, 0, sizeof(DBDecimalType) * COUNT5abfc2abe530);
DBStringType* d_aggr__c_phone;
hipMalloc(&d_aggr__c_phone, sizeof(DBStringType) * COUNT5abfc2abe530);
hipMemset(d_aggr__c_phone, 0, sizeof(DBStringType) * COUNT5abfc2abe530);
DBStringType* d_aggr__n_name;
hipMalloc(&d_aggr__n_name, sizeof(DBStringType) * COUNT5abfc2abe530);
hipMemset(d_aggr__n_name, 0, sizeof(DBStringType) * COUNT5abfc2abe530);
DBStringType* d_aggr__c_address;
hipMalloc(&d_aggr__c_address, sizeof(DBStringType) * COUNT5abfc2abe530);
hipMemset(d_aggr__c_address, 0, sizeof(DBStringType) * COUNT5abfc2abe530);
DBI32Type* d_KEY_5abfc2abe530customer__c_custkey;
hipMalloc(&d_KEY_5abfc2abe530customer__c_custkey, sizeof(DBI32Type) * COUNT5abfc2abe530);
hipMemset(d_KEY_5abfc2abe530customer__c_custkey, 0, sizeof(DBI32Type) * COUNT5abfc2abe530);
main_5abfc2a0b910<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5abfc2b014a0, d_BUF_5abfc2b03040, d_HT_5abfc2abe530.ref(cuco::find), d_HT_5abfc2b014a0.ref(cuco::for_each), d_HT_5abfc2b03040.ref(cuco::for_each), d_KEY_5abfc2abe530customer__c_custkey, d_aggr0__tmp_attr0, d_aggr__c_acctbal, d_aggr__c_address, d_aggr__c_comment, d_aggr__c_name, d_aggr__c_phone, d_aggr__n_name, d_customer__c_acctbal, d_customer__c_address, d_customer__c_comment, d_customer__c_custkey, d_customer__c_name, d_customer__c_nationkey, d_customer__c_phone, customer_size, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_nation__n_name);
//Materialize count
uint64_t* d_COUNT5abfc2ad1890;
hipMalloc(&d_COUNT5abfc2ad1890, sizeof(uint64_t));
hipMemset(d_COUNT5abfc2ad1890, 0, sizeof(uint64_t));
count_5abfc2b1d700<<<std::ceil((float)COUNT5abfc2abe530/32.), 32>>>(COUNT5abfc2abe530, d_COUNT5abfc2ad1890);
uint64_t COUNT5abfc2ad1890;
hipMemcpy(&COUNT5abfc2ad1890, d_COUNT5abfc2ad1890, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5abfc2ad1890);
//Materialize buffers
uint64_t* d_MAT_IDX5abfc2ad1890;
hipMalloc(&d_MAT_IDX5abfc2ad1890, sizeof(uint64_t));
hipMemset(d_MAT_IDX5abfc2ad1890, 0, sizeof(uint64_t));
auto MAT5abfc2ad1890customer__c_custkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5abfc2ad1890);
DBI32Type* d_MAT5abfc2ad1890customer__c_custkey;
hipMalloc(&d_MAT5abfc2ad1890customer__c_custkey, sizeof(DBI32Type) * COUNT5abfc2ad1890);
auto MAT5abfc2ad1890aggr__c_name = (DBStringType*)malloc(sizeof(DBStringType) * COUNT5abfc2ad1890);
DBStringType* d_MAT5abfc2ad1890aggr__c_name;
hipMalloc(&d_MAT5abfc2ad1890aggr__c_name, sizeof(DBStringType) * COUNT5abfc2ad1890);
auto MAT5abfc2ad1890aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5abfc2ad1890);
DBDecimalType* d_MAT5abfc2ad1890aggr0__tmp_attr0;
hipMalloc(&d_MAT5abfc2ad1890aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5abfc2ad1890);
auto MAT5abfc2ad1890aggr__c_acctbal = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5abfc2ad1890);
DBDecimalType* d_MAT5abfc2ad1890aggr__c_acctbal;
hipMalloc(&d_MAT5abfc2ad1890aggr__c_acctbal, sizeof(DBDecimalType) * COUNT5abfc2ad1890);
auto MAT5abfc2ad1890aggr__n_name = (DBStringType*)malloc(sizeof(DBStringType) * COUNT5abfc2ad1890);
DBStringType* d_MAT5abfc2ad1890aggr__n_name;
hipMalloc(&d_MAT5abfc2ad1890aggr__n_name, sizeof(DBStringType) * COUNT5abfc2ad1890);
auto MAT5abfc2ad1890aggr__c_address = (DBStringType*)malloc(sizeof(DBStringType) * COUNT5abfc2ad1890);
DBStringType* d_MAT5abfc2ad1890aggr__c_address;
hipMalloc(&d_MAT5abfc2ad1890aggr__c_address, sizeof(DBStringType) * COUNT5abfc2ad1890);
auto MAT5abfc2ad1890aggr__c_phone = (DBStringType*)malloc(sizeof(DBStringType) * COUNT5abfc2ad1890);
DBStringType* d_MAT5abfc2ad1890aggr__c_phone;
hipMalloc(&d_MAT5abfc2ad1890aggr__c_phone, sizeof(DBStringType) * COUNT5abfc2ad1890);
auto MAT5abfc2ad1890aggr__c_comment = (DBStringType*)malloc(sizeof(DBStringType) * COUNT5abfc2ad1890);
DBStringType* d_MAT5abfc2ad1890aggr__c_comment;
hipMalloc(&d_MAT5abfc2ad1890aggr__c_comment, sizeof(DBStringType) * COUNT5abfc2ad1890);
main_5abfc2b1d700<<<std::ceil((float)COUNT5abfc2abe530/32.), 32>>>(COUNT5abfc2abe530, d_MAT5abfc2ad1890aggr0__tmp_attr0, d_MAT5abfc2ad1890aggr__c_acctbal, d_MAT5abfc2ad1890aggr__c_address, d_MAT5abfc2ad1890aggr__c_comment, d_MAT5abfc2ad1890aggr__c_name, d_MAT5abfc2ad1890aggr__c_phone, d_MAT5abfc2ad1890aggr__n_name, d_MAT5abfc2ad1890customer__c_custkey, d_MAT_IDX5abfc2ad1890, d_aggr0__tmp_attr0, d_aggr__c_acctbal, d_aggr__c_address, d_aggr__c_comment, d_aggr__c_name, d_aggr__c_phone, d_aggr__n_name, d_KEY_5abfc2abe530customer__c_custkey);
hipFree(d_MAT_IDX5abfc2ad1890);
hipMemcpy(MAT5abfc2ad1890customer__c_custkey, d_MAT5abfc2ad1890customer__c_custkey, sizeof(DBI32Type) * COUNT5abfc2ad1890, hipMemcpyDeviceToHost);
hipMemcpy(MAT5abfc2ad1890aggr__c_name, d_MAT5abfc2ad1890aggr__c_name, sizeof(DBStringType) * COUNT5abfc2ad1890, hipMemcpyDeviceToHost);
hipMemcpy(MAT5abfc2ad1890aggr0__tmp_attr0, d_MAT5abfc2ad1890aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5abfc2ad1890, hipMemcpyDeviceToHost);
hipMemcpy(MAT5abfc2ad1890aggr__c_acctbal, d_MAT5abfc2ad1890aggr__c_acctbal, sizeof(DBDecimalType) * COUNT5abfc2ad1890, hipMemcpyDeviceToHost);
hipMemcpy(MAT5abfc2ad1890aggr__n_name, d_MAT5abfc2ad1890aggr__n_name, sizeof(DBStringType) * COUNT5abfc2ad1890, hipMemcpyDeviceToHost);
hipMemcpy(MAT5abfc2ad1890aggr__c_address, d_MAT5abfc2ad1890aggr__c_address, sizeof(DBStringType) * COUNT5abfc2ad1890, hipMemcpyDeviceToHost);
hipMemcpy(MAT5abfc2ad1890aggr__c_phone, d_MAT5abfc2ad1890aggr__c_phone, sizeof(DBStringType) * COUNT5abfc2ad1890, hipMemcpyDeviceToHost);
hipMemcpy(MAT5abfc2ad1890aggr__c_comment, d_MAT5abfc2ad1890aggr__c_comment, sizeof(DBStringType) * COUNT5abfc2ad1890, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5abfc2ad1890; i++) { std::cout << MAT5abfc2ad1890customer__c_custkey[i] << "\t";
std::cout << MAT5abfc2ad1890aggr__c_name[i] << "\t";
std::cout << MAT5abfc2ad1890aggr0__tmp_attr0[i] << "\t";
std::cout << MAT5abfc2ad1890aggr__c_acctbal[i] << "\t";
std::cout << MAT5abfc2ad1890aggr__n_name[i] << "\t";
std::cout << MAT5abfc2ad1890aggr__c_address[i] << "\t";
std::cout << MAT5abfc2ad1890aggr__c_phone[i] << "\t";
std::cout << MAT5abfc2ad1890aggr__c_comment[i] << "\t";
std::cout << std::endl; }
}