#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5811f646c7d0(uint64_t* COUNT5811f6560270, DBStringType* customer__c_mktsegment, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_mktsegment = customer__c_mktsegment[tid];
if (!(evaluatePredicate(reg_customer__c_mktsegment, "BUILDING", Predicate::eq))) return;
//Materialize count
atomicAdd((int*)COUNT5811f6560270, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5811f646c7d0(uint64_t* BUF_5811f6560270, uint64_t* BUF_IDX_5811f6560270, HASHTABLE_INSERT HT_5811f6560270, DBI32Type* customer__c_custkey, DBStringType* customer__c_mktsegment, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_mktsegment = customer__c_mktsegment[tid];
if (!(evaluatePredicate(reg_customer__c_mktsegment, "BUILDING", Predicate::eq))) return;
uint64_t KEY_5811f6560270 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5811f6560270 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_5811f6560270 = atomicAdd((int*)BUF_IDX_5811f6560270, 1);
HT_5811f6560270.insert(cuco::pair{KEY_5811f6560270, buf_idx_5811f6560270});
BUF_5811f6560270[buf_idx_5811f6560270 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_5811f64ef620(uint64_t* BUF_5811f6560270, uint64_t* COUNT5811f6564290, HASHTABLE_PROBE HT_5811f6560270, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 9204, Predicate::lt))) return;
uint64_t KEY_5811f6560270 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_5811f6560270 |= reg_orders__o_custkey;
//Probe Hash table
HT_5811f6560270.for_each(KEY_5811f6560270, [&] __device__ (auto const SLOT_5811f6560270) {

auto const [slot_first5811f6560270, slot_second5811f6560270] = SLOT_5811f6560270;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT5811f6564290, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_5811f64ef620(uint64_t* BUF_5811f6560270, uint64_t* BUF_5811f6564290, uint64_t* BUF_IDX_5811f6564290, HASHTABLE_PROBE HT_5811f6560270, HASHTABLE_INSERT HT_5811f6564290, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 9204, Predicate::lt))) return;
uint64_t KEY_5811f6560270 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_5811f6560270 |= reg_orders__o_custkey;
//Probe Hash table
HT_5811f6560270.for_each(KEY_5811f6560270, [&] __device__ (auto const SLOT_5811f6560270) {
auto const [slot_first5811f6560270, slot_second5811f6560270] = SLOT_5811f6560270;
if (!(true)) return;
uint64_t KEY_5811f6564290 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_5811f6564290 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_5811f6564290 = atomicAdd((int*)BUF_IDX_5811f6564290, 1);
HT_5811f6564290.insert(cuco::pair{KEY_5811f6564290, buf_idx_5811f6564290});
BUF_5811f6564290[buf_idx_5811f6564290 * 2 + 0] = BUF_5811f6560270[slot_second5811f6560270 * 1 + 0];
BUF_5811f6564290[buf_idx_5811f6564290 * 2 + 1] = tid;
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5811f656ca50(uint64_t* BUF_5811f6564290, HASHTABLE_INSERT HT_5811f651c720, HASHTABLE_PROBE HT_5811f6564290, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9204, Predicate::gt))) return;
uint64_t KEY_5811f6564290 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5811f6564290 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5811f6564290.for_each(KEY_5811f6564290, [&] __device__ (auto const SLOT_5811f6564290) {

auto const [slot_first5811f6564290, slot_second5811f6564290] = SLOT_5811f6564290;
if (!(true)) return;
uint64_t KEY_5811f651c720 = 0;

KEY_5811f651c720 |= reg_lineitem__l_orderkey;
//Create aggregation hash table
HT_5811f651c720.insert(cuco::pair{KEY_5811f651c720, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5811f656ca50(uint64_t* BUF_5811f6564290, HASHTABLE_FIND HT_5811f651c720, HASHTABLE_PROBE HT_5811f6564290, DBI32Type* KEY_5811f651c720lineitem__l_orderkey, DBDecimalType* aggr0__tmp_attr0, DBDateType* aggr__o_orderdate, DBI32Type* aggr__o_shippriority, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, size_t lineitem_size, DBDateType* orders__o_orderdate, DBI32Type* orders__o_shippriority) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9204, Predicate::gt))) return;
uint64_t KEY_5811f6564290 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_5811f6564290 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_5811f6564290.for_each(KEY_5811f6564290, [&] __device__ (auto const SLOT_5811f6564290) {
auto const [slot_first5811f6564290, slot_second5811f6564290] = SLOT_5811f6564290;
if (!(true)) return;
uint64_t KEY_5811f651c720 = 0;

KEY_5811f651c720 |= reg_lineitem__l_orderkey;
//Aggregate in hashtable
auto buf_idx_5811f651c720 = HT_5811f651c720.find(KEY_5811f651c720)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5811f651c720], reg_map0__tmp_attr1);
auto reg_orders__o_shippriority = orders__o_shippriority[BUF_5811f6564290[slot_second5811f6564290 * 2 + 1]];
aggregate_any(&aggr__o_shippriority[buf_idx_5811f651c720], reg_orders__o_shippriority);
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_5811f6564290[slot_second5811f6564290 * 2 + 1]];
aggregate_any(&aggr__o_orderdate[buf_idx_5811f651c720], reg_orders__o_orderdate);
KEY_5811f651c720lineitem__l_orderkey[buf_idx_5811f651c720] = reg_lineitem__l_orderkey;
});
}
__global__ void count_5811f6577500(uint64_t* COUNT5811f64fc400, size_t COUNT5811f651c720) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5811f651c720) return;
//Materialize count
atomicAdd((int*)COUNT5811f64fc400, 1);
}
__global__ void main_5811f6577500(size_t COUNT5811f651c720, DBDecimalType* MAT5811f64fc400aggr0__tmp_attr0, DBDateType* MAT5811f64fc400aggr__o_orderdate, DBI32Type* MAT5811f64fc400aggr__o_shippriority, DBI32Type* MAT5811f64fc400lineitem__l_orderkey, uint64_t* MAT_IDX5811f64fc400, DBDecimalType* aggr0__tmp_attr0, DBDateType* aggr__o_orderdate, DBI32Type* aggr__o_shippriority, DBI32Type* lineitem__l_orderkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5811f651c720) return;
//Materialize buffers
auto mat_idx5811f64fc400 = atomicAdd((int*)MAT_IDX5811f64fc400, 1);
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];
MAT5811f64fc400lineitem__l_orderkey[mat_idx5811f64fc400] = reg_lineitem__l_orderkey;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5811f64fc400aggr0__tmp_attr0[mat_idx5811f64fc400] = reg_aggr0__tmp_attr0;
auto reg_aggr__o_orderdate = aggr__o_orderdate[tid];
MAT5811f64fc400aggr__o_orderdate[mat_idx5811f64fc400] = reg_aggr__o_orderdate;
auto reg_aggr__o_shippriority = aggr__o_shippriority[tid];
MAT5811f64fc400aggr__o_shippriority[mat_idx5811f64fc400] = reg_aggr__o_shippriority;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT5811f6560270;
hipMalloc(&d_COUNT5811f6560270, sizeof(uint64_t));
hipMemset(d_COUNT5811f6560270, 0, sizeof(uint64_t));
count_5811f646c7d0<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT5811f6560270, d_customer__c_mktsegment, customer_size);
uint64_t COUNT5811f6560270;
hipMemcpy(&COUNT5811f6560270, d_COUNT5811f6560270, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5811f6560270);
// Insert hash table control;
uint64_t* d_BUF_IDX_5811f6560270;
hipMalloc(&d_BUF_IDX_5811f6560270, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5811f6560270, 0, sizeof(uint64_t));
uint64_t* d_BUF_5811f6560270;
hipMalloc(&d_BUF_5811f6560270, sizeof(uint64_t) * COUNT5811f6560270 * 1);
auto d_HT_5811f6560270 = cuco::experimental::static_multimap{ (int)COUNT5811f6560270*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5811f646c7d0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5811f6560270, d_BUF_IDX_5811f6560270, d_HT_5811f6560270.ref(cuco::insert), d_customer__c_custkey, d_customer__c_mktsegment, customer_size);
hipFree(d_BUF_IDX_5811f6560270);
//Materialize count
uint64_t* d_COUNT5811f6564290;
hipMalloc(&d_COUNT5811f6564290, sizeof(uint64_t));
hipMemset(d_COUNT5811f6564290, 0, sizeof(uint64_t));
count_5811f64ef620<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5811f6560270, d_COUNT5811f6564290, d_HT_5811f6560270.ref(cuco::for_each), d_orders__o_custkey, d_orders__o_orderdate, orders_size);
uint64_t COUNT5811f6564290;
hipMemcpy(&COUNT5811f6564290, d_COUNT5811f6564290, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5811f6564290);
// Insert hash table control;
uint64_t* d_BUF_IDX_5811f6564290;
hipMalloc(&d_BUF_IDX_5811f6564290, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5811f6564290, 0, sizeof(uint64_t));
uint64_t* d_BUF_5811f6564290;
hipMalloc(&d_BUF_5811f6564290, sizeof(uint64_t) * COUNT5811f6564290 * 2);
auto d_HT_5811f6564290 = cuco::experimental::static_multimap{ (int)COUNT5811f6564290*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5811f64ef620<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5811f6560270, d_BUF_5811f6564290, d_BUF_IDX_5811f6564290, d_HT_5811f6560270.ref(cuco::for_each), d_HT_5811f6564290.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
hipFree(d_BUF_IDX_5811f6564290);
//Create aggregation hash table
auto d_HT_5811f651c720 = cuco::static_map{ (int)355555*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5811f656ca50<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5811f6564290, d_HT_5811f651c720.ref(cuco::insert), d_HT_5811f6564290.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_shipdate, lineitem_size);
size_t COUNT5811f651c720 = d_HT_5811f651c720.size();
thrust::device_vector<int64_t> keys_5811f651c720(COUNT5811f651c720), vals_5811f651c720(COUNT5811f651c720);
d_HT_5811f651c720.retrieve_all(keys_5811f651c720.begin(), vals_5811f651c720.begin());
d_HT_5811f651c720.clear();
int64_t* raw_keys5811f651c720 = thrust::raw_pointer_cast(keys_5811f651c720.data());
insertKeys<<<std::ceil((float)COUNT5811f651c720/32.), 32>>>(raw_keys5811f651c720, d_HT_5811f651c720.ref(cuco::insert), COUNT5811f651c720);
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5811f651c720);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT5811f651c720);
DBI32Type* d_aggr__o_shippriority;
hipMalloc(&d_aggr__o_shippriority, sizeof(DBI32Type) * COUNT5811f651c720);
hipMemset(d_aggr__o_shippriority, 0, sizeof(DBI32Type) * COUNT5811f651c720);
DBDateType* d_aggr__o_orderdate;
hipMalloc(&d_aggr__o_orderdate, sizeof(DBDateType) * COUNT5811f651c720);
hipMemset(d_aggr__o_orderdate, 0, sizeof(DBDateType) * COUNT5811f651c720);
DBI32Type* d_KEY_5811f651c720lineitem__l_orderkey;
hipMalloc(&d_KEY_5811f651c720lineitem__l_orderkey, sizeof(DBI32Type) * COUNT5811f651c720);
hipMemset(d_KEY_5811f651c720lineitem__l_orderkey, 0, sizeof(DBI32Type) * COUNT5811f651c720);
main_5811f656ca50<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_5811f6564290, d_HT_5811f651c720.ref(cuco::find), d_HT_5811f6564290.ref(cuco::for_each), d_KEY_5811f651c720lineitem__l_orderkey, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, lineitem_size, d_orders__o_orderdate, d_orders__o_shippriority);
//Materialize count
uint64_t* d_COUNT5811f64fc400;
hipMalloc(&d_COUNT5811f64fc400, sizeof(uint64_t));
hipMemset(d_COUNT5811f64fc400, 0, sizeof(uint64_t));
count_5811f6577500<<<std::ceil((float)COUNT5811f651c720/32.), 32>>>(d_COUNT5811f64fc400, COUNT5811f651c720);
uint64_t COUNT5811f64fc400;
hipMemcpy(&COUNT5811f64fc400, d_COUNT5811f64fc400, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5811f64fc400);
//Materialize buffers
uint64_t* d_MAT_IDX5811f64fc400;
hipMalloc(&d_MAT_IDX5811f64fc400, sizeof(uint64_t));
hipMemset(d_MAT_IDX5811f64fc400, 0, sizeof(uint64_t));
auto MAT5811f64fc400lineitem__l_orderkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5811f64fc400);
DBI32Type* d_MAT5811f64fc400lineitem__l_orderkey;
hipMalloc(&d_MAT5811f64fc400lineitem__l_orderkey, sizeof(DBI32Type) * COUNT5811f64fc400);
auto MAT5811f64fc400aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT5811f64fc400);
DBDecimalType* d_MAT5811f64fc400aggr0__tmp_attr0;
hipMalloc(&d_MAT5811f64fc400aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5811f64fc400);
auto MAT5811f64fc400aggr__o_orderdate = (DBDateType*)malloc(sizeof(DBDateType) * COUNT5811f64fc400);
DBDateType* d_MAT5811f64fc400aggr__o_orderdate;
hipMalloc(&d_MAT5811f64fc400aggr__o_orderdate, sizeof(DBDateType) * COUNT5811f64fc400);
auto MAT5811f64fc400aggr__o_shippriority = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT5811f64fc400);
DBI32Type* d_MAT5811f64fc400aggr__o_shippriority;
hipMalloc(&d_MAT5811f64fc400aggr__o_shippriority, sizeof(DBI32Type) * COUNT5811f64fc400);
main_5811f6577500<<<std::ceil((float)COUNT5811f651c720/32.), 32>>>(COUNT5811f651c720, d_MAT5811f64fc400aggr0__tmp_attr0, d_MAT5811f64fc400aggr__o_orderdate, d_MAT5811f64fc400aggr__o_shippriority, d_MAT5811f64fc400lineitem__l_orderkey, d_MAT_IDX5811f64fc400, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_KEY_5811f651c720lineitem__l_orderkey);
hipFree(d_MAT_IDX5811f64fc400);
hipMemcpy(MAT5811f64fc400lineitem__l_orderkey, d_MAT5811f64fc400lineitem__l_orderkey, sizeof(DBI32Type) * COUNT5811f64fc400, hipMemcpyDeviceToHost);
hipMemcpy(MAT5811f64fc400aggr0__tmp_attr0, d_MAT5811f64fc400aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT5811f64fc400, hipMemcpyDeviceToHost);
hipMemcpy(MAT5811f64fc400aggr__o_orderdate, d_MAT5811f64fc400aggr__o_orderdate, sizeof(DBDateType) * COUNT5811f64fc400, hipMemcpyDeviceToHost);
hipMemcpy(MAT5811f64fc400aggr__o_shippriority, d_MAT5811f64fc400aggr__o_shippriority, sizeof(DBI32Type) * COUNT5811f64fc400, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5811f64fc400; i++) { std::cout << MAT5811f64fc400lineitem__l_orderkey[i] << "\t";
std::cout << MAT5811f64fc400aggr0__tmp_attr0[i] << "\t";
std::cout << MAT5811f64fc400aggr__o_orderdate[i] << "\t";
std::cout << MAT5811f64fc400aggr__o_shippriority[i] << "\t";
std::cout << std::endl; }
}