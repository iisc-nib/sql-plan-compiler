#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_59ce838df2c0(uint64_t* COUNT59ce839d2af0, DBStringType* customer__c_mktsegment, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_mktsegment = customer__c_mktsegment[tid];
if (!(evaluatePredicate(reg_customer__c_mktsegment, "BUILDING", Predicate::eq))) return;
//Materialize count
atomicAdd((int*)COUNT59ce839d2af0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_59ce838df2c0(uint64_t* BUF_59ce839d2af0, uint64_t* BUF_IDX_59ce839d2af0, HASHTABLE_INSERT HT_59ce839d2af0, DBI32Type* customer__c_custkey, DBStringType* customer__c_mktsegment, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
auto reg_customer__c_mktsegment = customer__c_mktsegment[tid];
if (!(evaluatePredicate(reg_customer__c_mktsegment, "BUILDING", Predicate::eq))) return;
uint64_t KEY_59ce839d2af0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_59ce839d2af0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_59ce839d2af0 = atomicAdd((int*)BUF_IDX_59ce839d2af0, 1);
HT_59ce839d2af0.insert(cuco::pair{KEY_59ce839d2af0, buf_idx_59ce839d2af0});
BUF_59ce839d2af0[buf_idx_59ce839d2af0 * 1 + 0] = tid;
}
template<typename HASHTABLE_PROBE>
__global__ void count_59ce83962030(uint64_t* BUF_59ce839d2af0, uint64_t* COUNT59ce839d4170, HASHTABLE_PROBE HT_59ce839d2af0, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 9204, Predicate::lt))) return;
uint64_t KEY_59ce839d2af0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_59ce839d2af0 |= reg_orders__o_custkey;
//Probe Hash table
HT_59ce839d2af0.for_each(KEY_59ce839d2af0, [&] __device__ (auto const SLOT_59ce839d2af0) {

auto const [slot_first59ce839d2af0, slot_second59ce839d2af0] = SLOT_59ce839d2af0;
if (!(true)) return;
//Materialize count
atomicAdd((int*)COUNT59ce839d4170, 1);
});
}
template<typename HASHTABLE_PROBE, typename HASHTABLE_INSERT>
__global__ void main_59ce83962030(uint64_t* BUF_59ce839d2af0, uint64_t* BUF_59ce839d4170, uint64_t* BUF_IDX_59ce839d4170, HASHTABLE_PROBE HT_59ce839d2af0, HASHTABLE_INSERT HT_59ce839d4170, DBI32Type* orders__o_custkey, DBDateType* orders__o_orderdate, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
auto reg_orders__o_orderdate = orders__o_orderdate[tid];
if (!(evaluatePredicate(reg_orders__o_orderdate, 9204, Predicate::lt))) return;
uint64_t KEY_59ce839d2af0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_59ce839d2af0 |= reg_orders__o_custkey;
//Probe Hash table
HT_59ce839d2af0.for_each(KEY_59ce839d2af0, [&] __device__ (auto const SLOT_59ce839d2af0) {
auto const [slot_first59ce839d2af0, slot_second59ce839d2af0] = SLOT_59ce839d2af0;
if (!(true)) return;
uint64_t KEY_59ce839d4170 = 0;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];

KEY_59ce839d4170 |= reg_orders__o_orderkey;
// Insert hash table kernel;
auto buf_idx_59ce839d4170 = atomicAdd((int*)BUF_IDX_59ce839d4170, 1);
HT_59ce839d4170.insert(cuco::pair{KEY_59ce839d4170, buf_idx_59ce839d4170});
BUF_59ce839d4170[buf_idx_59ce839d4170 * 2 + 0] = BUF_59ce839d2af0[slot_second59ce839d2af0 * 1 + 0];
BUF_59ce839d4170[buf_idx_59ce839d4170 * 2 + 1] = tid;
});
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_59ce839ded20(uint64_t* BUF_59ce839d4170, HASHTABLE_INSERT HT_59ce8398f540, HASHTABLE_PROBE HT_59ce839d4170, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, size_t lineitem_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9204, Predicate::gt))) return;
uint64_t KEY_59ce839d4170 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_59ce839d4170 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_59ce839d4170.for_each(KEY_59ce839d4170, [&] __device__ (auto const SLOT_59ce839d4170) {

auto const [slot_first59ce839d4170, slot_second59ce839d4170] = SLOT_59ce839d4170;
if (!(true)) return;
uint64_t KEY_59ce8398f540 = 0;

KEY_59ce8398f540 |= reg_lineitem__l_orderkey;
//Create aggregation hash table
HT_59ce8398f540.insert(cuco::pair{KEY_59ce8398f540, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_59ce839ded20(uint64_t* BUF_59ce839d4170, HASHTABLE_FIND HT_59ce8398f540, HASHTABLE_PROBE HT_59ce839d4170, DBI32Type* KEY_59ce8398f540lineitem__l_orderkey, DBDecimalType* aggr0__tmp_attr0, DBDateType* aggr__o_orderdate, DBI32Type* aggr__o_shippriority, DBDecimalType* lineitem__l_discount, DBDecimalType* lineitem__l_extendedprice, DBI32Type* lineitem__l_orderkey, DBDateType* lineitem__l_shipdate, size_t lineitem_size, DBDateType* orders__o_orderdate, DBI32Type* orders__o_shippriority) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= lineitem_size) return;
auto reg_lineitem__l_shipdate = lineitem__l_shipdate[tid];
if (!(evaluatePredicate(reg_lineitem__l_shipdate, 9204, Predicate::gt))) return;
uint64_t KEY_59ce839d4170 = 0;
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];

KEY_59ce839d4170 |= reg_lineitem__l_orderkey;
//Probe Hash table
HT_59ce839d4170.for_each(KEY_59ce839d4170, [&] __device__ (auto const SLOT_59ce839d4170) {
auto const [slot_first59ce839d4170, slot_second59ce839d4170] = SLOT_59ce839d4170;
if (!(true)) return;
uint64_t KEY_59ce8398f540 = 0;

KEY_59ce8398f540 |= reg_lineitem__l_orderkey;
//Aggregate in hashtable
auto buf_idx_59ce8398f540 = HT_59ce8398f540.find(KEY_59ce8398f540)->second;
auto reg_lineitem__l_discount = lineitem__l_discount[tid];
auto reg_lineitem__l_extendedprice = lineitem__l_extendedprice[tid];
auto reg_map0__tmp_attr1 = (reg_lineitem__l_extendedprice) * ((1) - (reg_lineitem__l_discount));
aggregate_sum(&aggr0__tmp_attr0[buf_idx_59ce8398f540], reg_map0__tmp_attr1);
auto reg_orders__o_shippriority = orders__o_shippriority[BUF_59ce839d4170[slot_second59ce839d4170 * 2 + 1]];
aggregate_any(&aggr__o_shippriority[buf_idx_59ce8398f540], reg_orders__o_shippriority);
auto reg_orders__o_orderdate = orders__o_orderdate[BUF_59ce839d4170[slot_second59ce839d4170 * 2 + 1]];
aggregate_any(&aggr__o_orderdate[buf_idx_59ce8398f540], reg_orders__o_orderdate);
KEY_59ce8398f540lineitem__l_orderkey[buf_idx_59ce8398f540] = reg_lineitem__l_orderkey;
});
}
__global__ void count_59ce839e9fa0(uint64_t* COUNT59ce8396ea00, size_t COUNT59ce8398f540) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT59ce8398f540) return;
//Materialize count
atomicAdd((int*)COUNT59ce8396ea00, 1);
}
__global__ void main_59ce839e9fa0(size_t COUNT59ce8398f540, DBDecimalType* MAT59ce8396ea00aggr0__tmp_attr0, DBDateType* MAT59ce8396ea00aggr__o_orderdate, DBI32Type* MAT59ce8396ea00aggr__o_shippriority, DBI32Type* MAT59ce8396ea00lineitem__l_orderkey, uint64_t* MAT_IDX59ce8396ea00, DBDecimalType* aggr0__tmp_attr0, DBDateType* aggr__o_orderdate, DBI32Type* aggr__o_shippriority, DBI32Type* lineitem__l_orderkey) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT59ce8398f540) return;
//Materialize buffers
auto mat_idx59ce8396ea00 = atomicAdd((int*)MAT_IDX59ce8396ea00, 1);
auto reg_lineitem__l_orderkey = lineitem__l_orderkey[tid];
MAT59ce8396ea00lineitem__l_orderkey[mat_idx59ce8396ea00] = reg_lineitem__l_orderkey;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT59ce8396ea00aggr0__tmp_attr0[mat_idx59ce8396ea00] = reg_aggr0__tmp_attr0;
auto reg_aggr__o_orderdate = aggr__o_orderdate[tid];
MAT59ce8396ea00aggr__o_orderdate[mat_idx59ce8396ea00] = reg_aggr__o_orderdate;
auto reg_aggr__o_shippriority = aggr__o_shippriority[tid];
MAT59ce8396ea00aggr__o_shippriority[mat_idx59ce8396ea00] = reg_aggr__o_shippriority;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT59ce839d2af0;
hipMalloc(&d_COUNT59ce839d2af0, sizeof(uint64_t));
hipMemset(d_COUNT59ce839d2af0, 0, sizeof(uint64_t));
count_59ce838df2c0<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT59ce839d2af0, d_customer__c_mktsegment, customer_size);
uint64_t COUNT59ce839d2af0;
hipMemcpy(&COUNT59ce839d2af0, d_COUNT59ce839d2af0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT59ce839d2af0);
// Insert hash table control;
uint64_t* d_BUF_IDX_59ce839d2af0;
hipMalloc(&d_BUF_IDX_59ce839d2af0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_59ce839d2af0, 0, sizeof(uint64_t));
uint64_t* d_BUF_59ce839d2af0;
hipMalloc(&d_BUF_59ce839d2af0, sizeof(uint64_t) * COUNT59ce839d2af0 * 1);
auto d_HT_59ce839d2af0 = cuco::experimental::static_multimap{ (int)COUNT59ce839d2af0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_59ce838df2c0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_59ce839d2af0, d_BUF_IDX_59ce839d2af0, d_HT_59ce839d2af0.ref(cuco::insert), d_customer__c_custkey, d_customer__c_mktsegment, customer_size);
hipFree(d_BUF_IDX_59ce839d2af0);
//Materialize count
uint64_t* d_COUNT59ce839d4170;
hipMalloc(&d_COUNT59ce839d4170, sizeof(uint64_t));
hipMemset(d_COUNT59ce839d4170, 0, sizeof(uint64_t));
count_59ce83962030<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_59ce839d2af0, d_COUNT59ce839d4170, d_HT_59ce839d2af0.ref(cuco::for_each), d_orders__o_custkey, d_orders__o_orderdate, orders_size);
uint64_t COUNT59ce839d4170;
hipMemcpy(&COUNT59ce839d4170, d_COUNT59ce839d4170, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT59ce839d4170);
// Insert hash table control;
uint64_t* d_BUF_IDX_59ce839d4170;
hipMalloc(&d_BUF_IDX_59ce839d4170, sizeof(uint64_t));
hipMemset(d_BUF_IDX_59ce839d4170, 0, sizeof(uint64_t));
uint64_t* d_BUF_59ce839d4170;
hipMalloc(&d_BUF_59ce839d4170, sizeof(uint64_t) * COUNT59ce839d4170 * 2);
auto d_HT_59ce839d4170 = cuco::experimental::static_multimap{ (int)COUNT59ce839d4170*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_59ce83962030<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_59ce839d2af0, d_BUF_59ce839d4170, d_BUF_IDX_59ce839d4170, d_HT_59ce839d2af0.ref(cuco::for_each), d_HT_59ce839d4170.ref(cuco::insert), d_orders__o_custkey, d_orders__o_orderdate, d_orders__o_orderkey, orders_size);
hipFree(d_BUF_IDX_59ce839d4170);
//Create aggregation hash table
auto d_HT_59ce8398f540 = cuco::static_map{ (int)355555*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_59ce839ded20<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_59ce839d4170, d_HT_59ce8398f540.ref(cuco::insert), d_HT_59ce839d4170.ref(cuco::for_each), d_lineitem__l_orderkey, d_lineitem__l_shipdate, lineitem_size);
size_t COUNT59ce8398f540 = d_HT_59ce8398f540.size();
thrust::device_vector<int64_t> keys_59ce8398f540(COUNT59ce8398f540), vals_59ce8398f540(COUNT59ce8398f540);
d_HT_59ce8398f540.retrieve_all(keys_59ce8398f540.begin(), vals_59ce8398f540.begin());
thrust::host_vector<int64_t> h_keys_59ce8398f540(COUNT59ce8398f540);
thrust::copy(keys_59ce8398f540.begin(), keys_59ce8398f540.end(), h_keys_59ce8398f540.begin());
thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_59ce8398f540(COUNT59ce8398f540);
for (int i=0; i < COUNT59ce8398f540; i++)
{actual_dict_59ce8398f540[i] = cuco::make_pair(h_keys_59ce8398f540[i], i);}
d_HT_59ce8398f540.clear();
d_HT_59ce8398f540.insert(actual_dict_59ce8398f540.begin(), actual_dict_59ce8398f540.end());
//Aggregate in hashtable
DBDecimalType* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT59ce8398f540);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBDecimalType) * COUNT59ce8398f540);
DBI32Type* d_aggr__o_shippriority;
hipMalloc(&d_aggr__o_shippriority, sizeof(DBI32Type) * COUNT59ce8398f540);
hipMemset(d_aggr__o_shippriority, 0, sizeof(DBI32Type) * COUNT59ce8398f540);
DBDateType* d_aggr__o_orderdate;
hipMalloc(&d_aggr__o_orderdate, sizeof(DBDateType) * COUNT59ce8398f540);
hipMemset(d_aggr__o_orderdate, 0, sizeof(DBDateType) * COUNT59ce8398f540);
DBI32Type* d_KEY_59ce8398f540lineitem__l_orderkey;
hipMalloc(&d_KEY_59ce8398f540lineitem__l_orderkey, sizeof(DBI32Type) * COUNT59ce8398f540);
hipMemset(d_KEY_59ce8398f540lineitem__l_orderkey, 0, sizeof(DBI32Type) * COUNT59ce8398f540);
main_59ce839ded20<<<std::ceil((float)lineitem_size/32.), 32>>>(d_BUF_59ce839d4170, d_HT_59ce8398f540.ref(cuco::find), d_HT_59ce839d4170.ref(cuco::for_each), d_KEY_59ce8398f540lineitem__l_orderkey, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_lineitem__l_discount, d_lineitem__l_extendedprice, d_lineitem__l_orderkey, d_lineitem__l_shipdate, lineitem_size, d_orders__o_orderdate, d_orders__o_shippriority);
//Materialize count
uint64_t* d_COUNT59ce8396ea00;
hipMalloc(&d_COUNT59ce8396ea00, sizeof(uint64_t));
hipMemset(d_COUNT59ce8396ea00, 0, sizeof(uint64_t));
count_59ce839e9fa0<<<std::ceil((float)COUNT59ce8398f540/32.), 32>>>(d_COUNT59ce8396ea00, COUNT59ce8398f540);
uint64_t COUNT59ce8396ea00;
hipMemcpy(&COUNT59ce8396ea00, d_COUNT59ce8396ea00, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT59ce8396ea00);
//Materialize buffers
uint64_t* d_MAT_IDX59ce8396ea00;
hipMalloc(&d_MAT_IDX59ce8396ea00, sizeof(uint64_t));
hipMemset(d_MAT_IDX59ce8396ea00, 0, sizeof(uint64_t));
auto MAT59ce8396ea00lineitem__l_orderkey = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT59ce8396ea00);
DBI32Type* d_MAT59ce8396ea00lineitem__l_orderkey;
hipMalloc(&d_MAT59ce8396ea00lineitem__l_orderkey, sizeof(DBI32Type) * COUNT59ce8396ea00);
auto MAT59ce8396ea00aggr0__tmp_attr0 = (DBDecimalType*)malloc(sizeof(DBDecimalType) * COUNT59ce8396ea00);
DBDecimalType* d_MAT59ce8396ea00aggr0__tmp_attr0;
hipMalloc(&d_MAT59ce8396ea00aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT59ce8396ea00);
auto MAT59ce8396ea00aggr__o_orderdate = (DBDateType*)malloc(sizeof(DBDateType) * COUNT59ce8396ea00);
DBDateType* d_MAT59ce8396ea00aggr__o_orderdate;
hipMalloc(&d_MAT59ce8396ea00aggr__o_orderdate, sizeof(DBDateType) * COUNT59ce8396ea00);
auto MAT59ce8396ea00aggr__o_shippriority = (DBI32Type*)malloc(sizeof(DBI32Type) * COUNT59ce8396ea00);
DBI32Type* d_MAT59ce8396ea00aggr__o_shippriority;
hipMalloc(&d_MAT59ce8396ea00aggr__o_shippriority, sizeof(DBI32Type) * COUNT59ce8396ea00);
main_59ce839e9fa0<<<std::ceil((float)COUNT59ce8398f540/32.), 32>>>(COUNT59ce8398f540, d_MAT59ce8396ea00aggr0__tmp_attr0, d_MAT59ce8396ea00aggr__o_orderdate, d_MAT59ce8396ea00aggr__o_shippriority, d_MAT59ce8396ea00lineitem__l_orderkey, d_MAT_IDX59ce8396ea00, d_aggr0__tmp_attr0, d_aggr__o_orderdate, d_aggr__o_shippriority, d_KEY_59ce8398f540lineitem__l_orderkey);
hipFree(d_MAT_IDX59ce8396ea00);
hipMemcpy(MAT59ce8396ea00lineitem__l_orderkey, d_MAT59ce8396ea00lineitem__l_orderkey, sizeof(DBI32Type) * COUNT59ce8396ea00, hipMemcpyDeviceToHost);
hipMemcpy(MAT59ce8396ea00aggr0__tmp_attr0, d_MAT59ce8396ea00aggr0__tmp_attr0, sizeof(DBDecimalType) * COUNT59ce8396ea00, hipMemcpyDeviceToHost);
hipMemcpy(MAT59ce8396ea00aggr__o_orderdate, d_MAT59ce8396ea00aggr__o_orderdate, sizeof(DBDateType) * COUNT59ce8396ea00, hipMemcpyDeviceToHost);
hipMemcpy(MAT59ce8396ea00aggr__o_shippriority, d_MAT59ce8396ea00aggr__o_shippriority, sizeof(DBI32Type) * COUNT59ce8396ea00, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT59ce8396ea00; i++) { std::cout << MAT59ce8396ea00lineitem__l_orderkey[i] << "\t";
std::cout << MAT59ce8396ea00aggr0__tmp_attr0[i] << "\t";
std::cout << MAT59ce8396ea00aggr__o_orderdate[i] << "\t";
std::cout << MAT59ce8396ea00aggr__o_shippriority[i] << "\t";
std::cout << std::endl; }
}