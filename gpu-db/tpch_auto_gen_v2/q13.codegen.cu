#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5906d56b92c0(uint64_t* COUNT5906d57a6c60, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
//Materialize count
atomicAdd((int*)COUNT5906d57a6c60, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5906d56b92c0(uint64_t* BUF_5906d57a6c60, uint64_t* BUF_IDX_5906d57a6c60, HASHTABLE_INSERT HT_5906d57a6c60, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_5906d57a6c60 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5906d57a6c60 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_5906d57a6c60 = atomicAdd((int*)BUF_IDX_5906d57a6c60, 1);
HT_5906d57a6c60.insert(cuco::pair{KEY_5906d57a6c60, buf_idx_5906d57a6c60});
BUF_5906d57a6c60[buf_idx_5906d57a6c60 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5906d573b0a0(uint64_t* BUF_5906d57a6c60, HASHTABLE_INSERT HT_5906d5762b30, HASHTABLE_PROBE HT_5906d57a6c60, DBI32Type* customer__c_custkey, DBI32Type* orders__o_custkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_5906d57a6c60 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_5906d57a6c60 |= reg_orders__o_custkey;
//Probe Hash table
HT_5906d57a6c60.for_each(KEY_5906d57a6c60, [&] __device__ (auto const SLOT_5906d57a6c60) {

auto const [slot_first5906d57a6c60, slot_second5906d57a6c60] = SLOT_5906d57a6c60;
if (!(true)) return;
uint64_t KEY_5906d5762b30 = 0;
auto reg_customer__c_custkey = customer__c_custkey[BUF_5906d57a6c60[slot_second5906d57a6c60 * 1 + 0]];

KEY_5906d5762b30 |= reg_customer__c_custkey;
//Create aggregation hash table
HT_5906d5762b30.insert(cuco::pair{KEY_5906d5762b30, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5906d573b0a0(uint64_t* BUF_5906d57a6c60, HASHTABLE_FIND HT_5906d5762b30, HASHTABLE_PROBE HT_5906d57a6c60, DBI32Type* KEY_5906d5762b30customer__c_custkey, DBI64Type* aggr0__tmp_attr0, DBI32Type* customer__c_custkey, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_5906d57a6c60 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_5906d57a6c60 |= reg_orders__o_custkey;
//Probe Hash table
HT_5906d57a6c60.for_each(KEY_5906d57a6c60, [&] __device__ (auto const SLOT_5906d57a6c60) {
auto const [slot_first5906d57a6c60, slot_second5906d57a6c60] = SLOT_5906d57a6c60;
if (!(true)) return;
uint64_t KEY_5906d5762b30 = 0;
auto reg_customer__c_custkey = customer__c_custkey[BUF_5906d57a6c60[slot_second5906d57a6c60 * 1 + 0]];

KEY_5906d5762b30 |= reg_customer__c_custkey;
//Aggregate in hashtable
auto buf_idx_5906d5762b30 = HT_5906d5762b30.find(KEY_5906d5762b30)->second;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5906d5762b30], 1);
KEY_5906d5762b30customer__c_custkey[buf_idx_5906d5762b30] = reg_customer__c_custkey;
});
}
template<typename HASHTABLE_INSERT>
__global__ void count_5906d57b04b0(size_t COUNT5906d5762b30, HASHTABLE_INSERT HT_5906d5763ea0, DBI64Type* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5906d5762b30) return;
uint64_t KEY_5906d5763ea0 = 0;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];

KEY_5906d5763ea0 |= reg_aggr0__tmp_attr0;
//Create aggregation hash table
HT_5906d5763ea0.insert(cuco::pair{KEY_5906d5763ea0, 1});
}
template<typename HASHTABLE_FIND>
__global__ void main_5906d57b04b0(size_t COUNT5906d5762b30, HASHTABLE_FIND HT_5906d5763ea0, DBI64Type* KEY_5906d5763ea0aggr0__tmp_attr0, DBI64Type* aggr0__tmp_attr0, DBI64Type* aggr1__tmp_attr1) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5906d5762b30) return;
uint64_t KEY_5906d5763ea0 = 0;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];

KEY_5906d5763ea0 |= reg_aggr0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_5906d5763ea0 = HT_5906d5763ea0.find(KEY_5906d5763ea0)->second;
aggregate_sum(&aggr1__tmp_attr1[buf_idx_5906d5763ea0], 1);
KEY_5906d5763ea0aggr0__tmp_attr0[buf_idx_5906d5763ea0] = reg_aggr0__tmp_attr0;
}
__global__ void count_5906d57b1ff0(size_t COUNT5906d5763ea0, uint64_t* COUNT5906d5777840) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5906d5763ea0) return;
//Materialize count
atomicAdd((int*)COUNT5906d5777840, 1);
}
__global__ void main_5906d57b1ff0(size_t COUNT5906d5763ea0, DBI64Type* MAT5906d5777840aggr0__tmp_attr0, DBI64Type* MAT5906d5777840aggr1__tmp_attr1, uint64_t* MAT_IDX5906d5777840, DBI64Type* aggr0__tmp_attr0, DBI64Type* aggr1__tmp_attr1) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5906d5763ea0) return;
//Materialize buffers
auto mat_idx5906d5777840 = atomicAdd((int*)MAT_IDX5906d5777840, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5906d5777840aggr0__tmp_attr0[mat_idx5906d5777840] = reg_aggr0__tmp_attr0;
auto reg_aggr1__tmp_attr1 = aggr1__tmp_attr1[tid];
MAT5906d5777840aggr1__tmp_attr1[mat_idx5906d5777840] = reg_aggr1__tmp_attr1;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT5906d57a6c60;
hipMalloc(&d_COUNT5906d57a6c60, sizeof(uint64_t));
hipMemset(d_COUNT5906d57a6c60, 0, sizeof(uint64_t));
count_5906d56b92c0<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT5906d57a6c60, customer_size);
uint64_t COUNT5906d57a6c60;
hipMemcpy(&COUNT5906d57a6c60, d_COUNT5906d57a6c60, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5906d57a6c60);
// Insert hash table control;
uint64_t* d_BUF_IDX_5906d57a6c60;
hipMalloc(&d_BUF_IDX_5906d57a6c60, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5906d57a6c60, 0, sizeof(uint64_t));
uint64_t* d_BUF_5906d57a6c60;
hipMalloc(&d_BUF_5906d57a6c60, sizeof(uint64_t) * COUNT5906d57a6c60 * 1);
auto d_HT_5906d57a6c60 = cuco::experimental::static_multimap{ (int)COUNT5906d57a6c60*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5906d56b92c0<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5906d57a6c60, d_BUF_IDX_5906d57a6c60, d_HT_5906d57a6c60.ref(cuco::insert), d_customer__c_custkey, customer_size);
hipFree(d_BUF_IDX_5906d57a6c60);
//Create aggregation hash table
auto d_HT_5906d5762b30 = cuco::static_map{ (int)1500000*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5906d573b0a0<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5906d57a6c60, d_HT_5906d5762b30.ref(cuco::insert), d_HT_5906d57a6c60.ref(cuco::for_each), d_customer__c_custkey, d_orders__o_custkey, orders_size);
size_t COUNT5906d5762b30 = d_HT_5906d5762b30.size();
thrust::device_vector<int64_t> keys_5906d5762b30(COUNT5906d5762b30), vals_5906d5762b30(COUNT5906d5762b30);
d_HT_5906d5762b30.retrieve_all(keys_5906d5762b30.begin(), vals_5906d5762b30.begin());
thrust::host_vector<int64_t> h_keys_5906d5762b30(COUNT5906d5762b30);
thrust::copy(keys_5906d5762b30.begin(), keys_5906d5762b30.end(), h_keys_5906d5762b30.begin());
thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_5906d5762b30(COUNT5906d5762b30);
for (int i=0; i < COUNT5906d5762b30; i++)
{actual_dict_5906d5762b30[i] = cuco::make_pair(h_keys_5906d5762b30[i], i);}
d_HT_5906d5762b30.clear();
d_HT_5906d5762b30.insert(actual_dict_5906d5762b30.begin(), actual_dict_5906d5762b30.end());
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT5906d5762b30);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT5906d5762b30);
DBI32Type* d_KEY_5906d5762b30customer__c_custkey;
hipMalloc(&d_KEY_5906d5762b30customer__c_custkey, sizeof(DBI32Type) * COUNT5906d5762b30);
hipMemset(d_KEY_5906d5762b30customer__c_custkey, 0, sizeof(DBI32Type) * COUNT5906d5762b30);
main_5906d573b0a0<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5906d57a6c60, d_HT_5906d5762b30.ref(cuco::find), d_HT_5906d57a6c60.ref(cuco::for_each), d_KEY_5906d5762b30customer__c_custkey, d_aggr0__tmp_attr0, d_customer__c_custkey, d_orders__o_custkey, d_orders__o_orderkey, orders_size);
//Create aggregation hash table
auto d_HT_5906d5763ea0 = cuco::static_map{ (int)1500000*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5906d57b04b0<<<std::ceil((float)COUNT5906d5762b30/32.), 32>>>(COUNT5906d5762b30, d_HT_5906d5763ea0.ref(cuco::insert), d_aggr0__tmp_attr0);
size_t COUNT5906d5763ea0 = d_HT_5906d5763ea0.size();
thrust::device_vector<int64_t> keys_5906d5763ea0(COUNT5906d5763ea0), vals_5906d5763ea0(COUNT5906d5763ea0);
d_HT_5906d5763ea0.retrieve_all(keys_5906d5763ea0.begin(), vals_5906d5763ea0.begin());
thrust::host_vector<int64_t> h_keys_5906d5763ea0(COUNT5906d5763ea0);
thrust::copy(keys_5906d5763ea0.begin(), keys_5906d5763ea0.end(), h_keys_5906d5763ea0.begin());
thrust::host_vector<cuco::pair<int64_t, int64_t>> actual_dict_5906d5763ea0(COUNT5906d5763ea0);
for (int i=0; i < COUNT5906d5763ea0; i++)
{actual_dict_5906d5763ea0[i] = cuco::make_pair(h_keys_5906d5763ea0[i], i);}
d_HT_5906d5763ea0.clear();
d_HT_5906d5763ea0.insert(actual_dict_5906d5763ea0.begin(), actual_dict_5906d5763ea0.end());
//Aggregate in hashtable
DBI64Type* d_aggr1__tmp_attr1;
hipMalloc(&d_aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT5906d5763ea0);
hipMemset(d_aggr1__tmp_attr1, 0, sizeof(DBI64Type) * COUNT5906d5763ea0);
DBI64Type* d_KEY_5906d5763ea0aggr0__tmp_attr0;
hipMalloc(&d_KEY_5906d5763ea0aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT5906d5763ea0);
hipMemset(d_KEY_5906d5763ea0aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT5906d5763ea0);
main_5906d57b04b0<<<std::ceil((float)COUNT5906d5762b30/32.), 32>>>(COUNT5906d5762b30, d_HT_5906d5763ea0.ref(cuco::find), d_KEY_5906d5763ea0aggr0__tmp_attr0, d_aggr0__tmp_attr0, d_aggr1__tmp_attr1);
//Materialize count
uint64_t* d_COUNT5906d5777840;
hipMalloc(&d_COUNT5906d5777840, sizeof(uint64_t));
hipMemset(d_COUNT5906d5777840, 0, sizeof(uint64_t));
count_5906d57b1ff0<<<std::ceil((float)COUNT5906d5763ea0/32.), 32>>>(COUNT5906d5763ea0, d_COUNT5906d5777840);
uint64_t COUNT5906d5777840;
hipMemcpy(&COUNT5906d5777840, d_COUNT5906d5777840, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5906d5777840);
//Materialize buffers
uint64_t* d_MAT_IDX5906d5777840;
hipMalloc(&d_MAT_IDX5906d5777840, sizeof(uint64_t));
hipMemset(d_MAT_IDX5906d5777840, 0, sizeof(uint64_t));
auto MAT5906d5777840aggr0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT5906d5777840);
DBI64Type* d_MAT5906d5777840aggr0__tmp_attr0;
hipMalloc(&d_MAT5906d5777840aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT5906d5777840);
auto MAT5906d5777840aggr1__tmp_attr1 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT5906d5777840);
DBI64Type* d_MAT5906d5777840aggr1__tmp_attr1;
hipMalloc(&d_MAT5906d5777840aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT5906d5777840);
main_5906d57b1ff0<<<std::ceil((float)COUNT5906d5763ea0/32.), 32>>>(COUNT5906d5763ea0, d_MAT5906d5777840aggr0__tmp_attr0, d_MAT5906d5777840aggr1__tmp_attr1, d_MAT_IDX5906d5777840, d_KEY_5906d5763ea0aggr0__tmp_attr0, d_aggr1__tmp_attr1);
hipFree(d_MAT_IDX5906d5777840);
hipMemcpy(MAT5906d5777840aggr0__tmp_attr0, d_MAT5906d5777840aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT5906d5777840, hipMemcpyDeviceToHost);
hipMemcpy(MAT5906d5777840aggr1__tmp_attr1, d_MAT5906d5777840aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT5906d5777840, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5906d5777840; i++) { std::cout << MAT5906d5777840aggr0__tmp_attr0[i] << "\t";
std::cout << MAT5906d5777840aggr1__tmp_attr1[i] << "\t";
std::cout << std::endl; }
}