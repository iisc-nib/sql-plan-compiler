#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_56f0025ab110(uint64_t* COUNT56f0025a6fd0, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
//Materialize count
atomicAdd((int*)COUNT56f0025a6fd0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_56f0025ab110(uint64_t* BUF_56f0025a6fd0, uint64_t* BUF_IDX_56f0025a6fd0, HASHTABLE_INSERT HT_56f0025a6fd0, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_56f0025a6fd0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_56f0025a6fd0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_56f0025a6fd0 = atomicAdd((int*)BUF_IDX_56f0025a6fd0, 1);
HT_56f0025a6fd0.insert(cuco::pair{KEY_56f0025a6fd0, buf_idx_56f0025a6fd0});
BUF_56f0025a6fd0[buf_idx_56f0025a6fd0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_56f0025abe20(uint64_t* BUF_56f0025a6fd0, HASHTABLE_INSERT HT_56f002565020, HASHTABLE_PROBE HT_56f0025a6fd0, DBI32Type* customer__c_custkey, DBI32Type* orders__o_custkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_56f0025a6fd0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_56f0025a6fd0 |= reg_orders__o_custkey;
//Probe Hash table
HT_56f0025a6fd0.for_each(KEY_56f0025a6fd0, [&] __device__ (auto const SLOT_56f0025a6fd0) {

auto const [slot_first56f0025a6fd0, slot_second56f0025a6fd0] = SLOT_56f0025a6fd0;
if (!(true)) return;
uint64_t KEY_56f002565020 = 0;
auto reg_customer__c_custkey = customer__c_custkey[BUF_56f0025a6fd0[slot_second56f0025a6fd0 * 1 + 0]];

KEY_56f002565020 |= reg_customer__c_custkey;
//Create aggregation hash table
HT_56f002565020.insert(cuco::pair{KEY_56f002565020, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_56f0025abe20(uint64_t* BUF_56f0025a6fd0, HASHTABLE_FIND HT_56f002565020, HASHTABLE_PROBE HT_56f0025a6fd0, DBI32Type* KEY_56f002565020customer__c_custkey, DBI64Type* aggr0__tmp_attr0, DBI32Type* customer__c_custkey, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_56f0025a6fd0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_56f0025a6fd0 |= reg_orders__o_custkey;
//Probe Hash table
HT_56f0025a6fd0.for_each(KEY_56f0025a6fd0, [&] __device__ (auto const SLOT_56f0025a6fd0) {
auto const [slot_first56f0025a6fd0, slot_second56f0025a6fd0] = SLOT_56f0025a6fd0;
if (!(true)) return;
uint64_t KEY_56f002565020 = 0;
auto reg_customer__c_custkey = customer__c_custkey[BUF_56f0025a6fd0[slot_second56f0025a6fd0 * 1 + 0]];

KEY_56f002565020 |= reg_customer__c_custkey;
//Aggregate in hashtable
auto buf_idx_56f002565020 = HT_56f002565020.find(KEY_56f002565020)->second;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_56f002565020], 1);
KEY_56f002565020customer__c_custkey[buf_idx_56f002565020] = reg_customer__c_custkey;
});
}
template<typename HASHTABLE_INSERT>
__global__ void count_56f0025b1760(size_t COUNT56f002565020, HASHTABLE_INSERT HT_56f002566800, DBI64Type* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT56f002565020) return;
uint64_t KEY_56f002566800 = 0;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];

KEY_56f002566800 |= reg_aggr0__tmp_attr0;
//Create aggregation hash table
HT_56f002566800.insert(cuco::pair{KEY_56f002566800, 1});
}
template<typename HASHTABLE_FIND>
__global__ void main_56f0025b1760(size_t COUNT56f002565020, HASHTABLE_FIND HT_56f002566800, DBI64Type* KEY_56f002566800aggr0__tmp_attr0, DBI64Type* aggr0__tmp_attr0, DBI64Type* aggr1__tmp_attr1) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT56f002565020) return;
uint64_t KEY_56f002566800 = 0;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];

KEY_56f002566800 |= reg_aggr0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_56f002566800 = HT_56f002566800.find(KEY_56f002566800)->second;
aggregate_sum(&aggr1__tmp_attr1[buf_idx_56f002566800], 1);
KEY_56f002566800aggr0__tmp_attr0[buf_idx_56f002566800] = reg_aggr0__tmp_attr0;
}
__global__ void count_56f0025b32b0(size_t COUNT56f002566800, uint64_t* COUNT56f0025798c0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT56f002566800) return;
//Materialize count
atomicAdd((int*)COUNT56f0025798c0, 1);
}
__global__ void main_56f0025b32b0(size_t COUNT56f002566800, DBI64Type* MAT56f0025798c0aggr0__tmp_attr0, DBI64Type* MAT56f0025798c0aggr1__tmp_attr1, uint64_t* MAT_IDX56f0025798c0, DBI64Type* aggr0__tmp_attr0, DBI64Type* aggr1__tmp_attr1) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT56f002566800) return;
//Materialize buffers
auto mat_idx56f0025798c0 = atomicAdd((int*)MAT_IDX56f0025798c0, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT56f0025798c0aggr0__tmp_attr0[mat_idx56f0025798c0] = reg_aggr0__tmp_attr0;
auto reg_aggr1__tmp_attr1 = aggr1__tmp_attr1[tid];
MAT56f0025798c0aggr1__tmp_attr1[mat_idx56f0025798c0] = reg_aggr1__tmp_attr1;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT56f0025a6fd0;
hipMalloc(&d_COUNT56f0025a6fd0, sizeof(uint64_t));
hipMemset(d_COUNT56f0025a6fd0, 0, sizeof(uint64_t));
count_56f0025ab110<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT56f0025a6fd0, customer_size);
uint64_t COUNT56f0025a6fd0;
hipMemcpy(&COUNT56f0025a6fd0, d_COUNT56f0025a6fd0, sizeof(uint64_t), hipMemcpyDeviceToHost);
// Insert hash table control;
uint64_t* d_BUF_IDX_56f0025a6fd0;
hipMalloc(&d_BUF_IDX_56f0025a6fd0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_56f0025a6fd0, 0, sizeof(uint64_t));
uint64_t* d_BUF_56f0025a6fd0;
hipMalloc(&d_BUF_56f0025a6fd0, sizeof(uint64_t) * COUNT56f0025a6fd0 * 1);
auto d_HT_56f0025a6fd0 = cuco::experimental::static_multimap{ (int)COUNT56f0025a6fd0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_56f0025ab110<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_56f0025a6fd0, d_BUF_IDX_56f0025a6fd0, d_HT_56f0025a6fd0.ref(cuco::insert), d_customer__c_custkey, customer_size);
//Create aggregation hash table
auto d_HT_56f002565020 = cuco::static_map{ (int)1500000*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_56f0025abe20<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_56f0025a6fd0, d_HT_56f002565020.ref(cuco::insert), d_HT_56f0025a6fd0.ref(cuco::for_each), d_customer__c_custkey, d_orders__o_custkey, orders_size);
size_t COUNT56f002565020 = d_HT_56f002565020.size();
thrust::device_vector<int64_t> keys_56f002565020(COUNT56f002565020), vals_56f002565020(COUNT56f002565020);
d_HT_56f002565020.retrieve_all(keys_56f002565020.begin(), vals_56f002565020.begin());
d_HT_56f002565020.clear();
int64_t* raw_keys56f002565020 = thrust::raw_pointer_cast(keys_56f002565020.data());
insertKeys<<<std::ceil((float)COUNT56f002565020/32.), 32>>>(raw_keys56f002565020, d_HT_56f002565020.ref(cuco::insert), COUNT56f002565020);
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT56f002565020);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT56f002565020);
DBI32Type* d_KEY_56f002565020customer__c_custkey;
hipMalloc(&d_KEY_56f002565020customer__c_custkey, sizeof(DBI32Type) * COUNT56f002565020);
hipMemset(d_KEY_56f002565020customer__c_custkey, 0, sizeof(DBI32Type) * COUNT56f002565020);
main_56f0025abe20<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_56f0025a6fd0, d_HT_56f002565020.ref(cuco::find), d_HT_56f0025a6fd0.ref(cuco::for_each), d_KEY_56f002565020customer__c_custkey, d_aggr0__tmp_attr0, d_customer__c_custkey, d_orders__o_custkey, d_orders__o_orderkey, orders_size);
//Create aggregation hash table
auto d_HT_56f002566800 = cuco::static_map{ (int)1500000*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_56f0025b1760<<<std::ceil((float)COUNT56f002565020/32.), 32>>>(COUNT56f002565020, d_HT_56f002566800.ref(cuco::insert), d_aggr0__tmp_attr0);
size_t COUNT56f002566800 = d_HT_56f002566800.size();
thrust::device_vector<int64_t> keys_56f002566800(COUNT56f002566800), vals_56f002566800(COUNT56f002566800);
d_HT_56f002566800.retrieve_all(keys_56f002566800.begin(), vals_56f002566800.begin());
d_HT_56f002566800.clear();
int64_t* raw_keys56f002566800 = thrust::raw_pointer_cast(keys_56f002566800.data());
insertKeys<<<std::ceil((float)COUNT56f002566800/32.), 32>>>(raw_keys56f002566800, d_HT_56f002566800.ref(cuco::insert), COUNT56f002566800);
//Aggregate in hashtable
DBI64Type* d_aggr1__tmp_attr1;
hipMalloc(&d_aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT56f002566800);
hipMemset(d_aggr1__tmp_attr1, 0, sizeof(DBI64Type) * COUNT56f002566800);
DBI64Type* d_KEY_56f002566800aggr0__tmp_attr0;
hipMalloc(&d_KEY_56f002566800aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT56f002566800);
hipMemset(d_KEY_56f002566800aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT56f002566800);
main_56f0025b1760<<<std::ceil((float)COUNT56f002565020/32.), 32>>>(COUNT56f002565020, d_HT_56f002566800.ref(cuco::find), d_KEY_56f002566800aggr0__tmp_attr0, d_aggr0__tmp_attr0, d_aggr1__tmp_attr1);
//Materialize count
uint64_t* d_COUNT56f0025798c0;
hipMalloc(&d_COUNT56f0025798c0, sizeof(uint64_t));
hipMemset(d_COUNT56f0025798c0, 0, sizeof(uint64_t));
count_56f0025b32b0<<<std::ceil((float)COUNT56f002566800/32.), 32>>>(COUNT56f002566800, d_COUNT56f0025798c0);
uint64_t COUNT56f0025798c0;
hipMemcpy(&COUNT56f0025798c0, d_COUNT56f0025798c0, sizeof(uint64_t), hipMemcpyDeviceToHost);
//Materialize buffers
uint64_t* d_MAT_IDX56f0025798c0;
hipMalloc(&d_MAT_IDX56f0025798c0, sizeof(uint64_t));
hipMemset(d_MAT_IDX56f0025798c0, 0, sizeof(uint64_t));
auto MAT56f0025798c0aggr0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT56f0025798c0);
DBI64Type* d_MAT56f0025798c0aggr0__tmp_attr0;
hipMalloc(&d_MAT56f0025798c0aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT56f0025798c0);
auto MAT56f0025798c0aggr1__tmp_attr1 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT56f0025798c0);
DBI64Type* d_MAT56f0025798c0aggr1__tmp_attr1;
hipMalloc(&d_MAT56f0025798c0aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT56f0025798c0);
main_56f0025b32b0<<<std::ceil((float)COUNT56f002566800/32.), 32>>>(COUNT56f002566800, d_MAT56f0025798c0aggr0__tmp_attr0, d_MAT56f0025798c0aggr1__tmp_attr1, d_MAT_IDX56f0025798c0, d_KEY_56f002566800aggr0__tmp_attr0, d_aggr1__tmp_attr1);
hipMemcpy(MAT56f0025798c0aggr0__tmp_attr0, d_MAT56f0025798c0aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT56f0025798c0, hipMemcpyDeviceToHost);
hipMemcpy(MAT56f0025798c0aggr1__tmp_attr1, d_MAT56f0025798c0aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT56f0025798c0, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT56f0025798c0; i++) { std::cout << MAT56f0025798c0aggr0__tmp_attr0[i] << "\t";
std::cout << MAT56f0025798c0aggr1__tmp_attr1[i] << "\t";
std::cout << std::endl; }
hipFree(d_BUF_56f0025a6fd0);
hipFree(d_BUF_IDX_56f0025a6fd0);
hipFree(d_COUNT56f0025a6fd0);
hipFree(d_KEY_56f002565020customer__c_custkey);
hipFree(d_aggr0__tmp_attr0);
hipFree(d_KEY_56f002566800aggr0__tmp_attr0);
hipFree(d_aggr1__tmp_attr1);
hipFree(d_COUNT56f0025798c0);
hipFree(d_MAT56f0025798c0aggr0__tmp_attr0);
hipFree(d_MAT56f0025798c0aggr1__tmp_attr1);
hipFree(d_MAT_IDX56f0025798c0);
free(MAT56f0025798c0aggr0__tmp_attr0);
free(MAT56f0025798c0aggr1__tmp_attr1);
}