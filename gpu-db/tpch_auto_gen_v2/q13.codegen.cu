#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <cuco/static_multimap.cuh>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "cudautils.cuh"
#include "db_types.h"
#include "dbruntime.h"
__global__ void count_5b84ec34b570(uint64_t* COUNT5b84ec4372d0, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
//Materialize count
atomicAdd((int*)COUNT5b84ec4372d0, 1);
}
template<typename HASHTABLE_INSERT>
__global__ void main_5b84ec34b570(uint64_t* BUF_5b84ec4372d0, uint64_t* BUF_IDX_5b84ec4372d0, HASHTABLE_INSERT HT_5b84ec4372d0, DBI32Type* customer__c_custkey, size_t customer_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= customer_size) return;
uint64_t KEY_5b84ec4372d0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[tid];

KEY_5b84ec4372d0 |= reg_customer__c_custkey;
// Insert hash table kernel;
auto buf_idx_5b84ec4372d0 = atomicAdd((int*)BUF_IDX_5b84ec4372d0, 1);
HT_5b84ec4372d0.insert(cuco::pair{KEY_5b84ec4372d0, buf_idx_5b84ec4372d0});
BUF_5b84ec4372d0[buf_idx_5b84ec4372d0 * 1 + 0] = tid;
}
template<typename HASHTABLE_INSERT, typename HASHTABLE_PROBE>
__global__ void count_5b84ec3cd3d0(uint64_t* BUF_5b84ec4372d0, HASHTABLE_INSERT HT_5b84ec3f5bb0, HASHTABLE_PROBE HT_5b84ec4372d0, DBI32Type* customer__c_custkey, DBI32Type* orders__o_custkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_5b84ec4372d0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_5b84ec4372d0 |= reg_orders__o_custkey;
//Probe Hash table
HT_5b84ec4372d0.for_each(KEY_5b84ec4372d0, [&] __device__ (auto const SLOT_5b84ec4372d0) {

auto const [slot_first5b84ec4372d0, slot_second5b84ec4372d0] = SLOT_5b84ec4372d0;
if (!(true)) return;
uint64_t KEY_5b84ec3f5bb0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[BUF_5b84ec4372d0[slot_second5b84ec4372d0 * 1 + 0]];

KEY_5b84ec3f5bb0 |= reg_customer__c_custkey;
//Create aggregation hash table
HT_5b84ec3f5bb0.insert(cuco::pair{KEY_5b84ec3f5bb0, 1});
});
}
template<typename HASHTABLE_FIND, typename HASHTABLE_PROBE>
__global__ void main_5b84ec3cd3d0(uint64_t* BUF_5b84ec4372d0, HASHTABLE_FIND HT_5b84ec3f5bb0, HASHTABLE_PROBE HT_5b84ec4372d0, DBI32Type* KEY_5b84ec3f5bb0customer__c_custkey, DBI64Type* aggr0__tmp_attr0, DBI32Type* customer__c_custkey, DBI32Type* orders__o_custkey, DBI32Type* orders__o_orderkey, size_t orders_size) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= orders_size) return;
uint64_t KEY_5b84ec4372d0 = 0;
auto reg_orders__o_custkey = orders__o_custkey[tid];

KEY_5b84ec4372d0 |= reg_orders__o_custkey;
//Probe Hash table
HT_5b84ec4372d0.for_each(KEY_5b84ec4372d0, [&] __device__ (auto const SLOT_5b84ec4372d0) {
auto const [slot_first5b84ec4372d0, slot_second5b84ec4372d0] = SLOT_5b84ec4372d0;
if (!(true)) return;
uint64_t KEY_5b84ec3f5bb0 = 0;
auto reg_customer__c_custkey = customer__c_custkey[BUF_5b84ec4372d0[slot_second5b84ec4372d0 * 1 + 0]];

KEY_5b84ec3f5bb0 |= reg_customer__c_custkey;
//Aggregate in hashtable
auto buf_idx_5b84ec3f5bb0 = HT_5b84ec3f5bb0.find(KEY_5b84ec3f5bb0)->second;
auto reg_orders__o_orderkey = orders__o_orderkey[tid];
aggregate_sum(&aggr0__tmp_attr0[buf_idx_5b84ec3f5bb0], 1);
KEY_5b84ec3f5bb0customer__c_custkey[buf_idx_5b84ec3f5bb0] = reg_customer__c_custkey;
});
}
template<typename HASHTABLE_INSERT>
__global__ void count_5b84ec442060(size_t COUNT5b84ec3f5bb0, HASHTABLE_INSERT HT_5b84ec3f61d0, DBI64Type* aggr0__tmp_attr0) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5b84ec3f5bb0) return;
uint64_t KEY_5b84ec3f61d0 = 0;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];

KEY_5b84ec3f61d0 |= reg_aggr0__tmp_attr0;
//Create aggregation hash table
HT_5b84ec3f61d0.insert(cuco::pair{KEY_5b84ec3f61d0, 1});
}
template<typename HASHTABLE_FIND>
__global__ void main_5b84ec442060(size_t COUNT5b84ec3f5bb0, HASHTABLE_FIND HT_5b84ec3f61d0, DBI64Type* KEY_5b84ec3f61d0aggr0__tmp_attr0, DBI64Type* aggr0__tmp_attr0, DBI64Type* aggr1__tmp_attr1) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5b84ec3f5bb0) return;
uint64_t KEY_5b84ec3f61d0 = 0;
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];

KEY_5b84ec3f61d0 |= reg_aggr0__tmp_attr0;
//Aggregate in hashtable
auto buf_idx_5b84ec3f61d0 = HT_5b84ec3f61d0.find(KEY_5b84ec3f61d0)->second;
aggregate_sum(&aggr1__tmp_attr1[buf_idx_5b84ec3f61d0], 1);
KEY_5b84ec3f61d0aggr0__tmp_attr0[buf_idx_5b84ec3f61d0] = reg_aggr0__tmp_attr0;
}
__global__ void count_5b84ec443a60(size_t COUNT5b84ec3f61d0, uint64_t* COUNT5b84ec409290) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5b84ec3f61d0) return;
//Materialize count
atomicAdd((int*)COUNT5b84ec409290, 1);
}
__global__ void main_5b84ec443a60(size_t COUNT5b84ec3f61d0, DBI64Type* MAT5b84ec409290aggr0__tmp_attr0, DBI64Type* MAT5b84ec409290aggr1__tmp_attr1, uint64_t* MAT_IDX5b84ec409290, DBI64Type* aggr0__tmp_attr0, DBI64Type* aggr1__tmp_attr1) {
size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
if (tid >= COUNT5b84ec3f61d0) return;
//Materialize buffers
auto mat_idx5b84ec409290 = atomicAdd((int*)MAT_IDX5b84ec409290, 1);
auto reg_aggr0__tmp_attr0 = aggr0__tmp_attr0[tid];
MAT5b84ec409290aggr0__tmp_attr0[mat_idx5b84ec409290] = reg_aggr0__tmp_attr0;
auto reg_aggr1__tmp_attr1 = aggr1__tmp_attr1[tid];
MAT5b84ec409290aggr1__tmp_attr1[mat_idx5b84ec409290] = reg_aggr1__tmp_attr1;
}
extern "C" void control( DBI32Type* d_nation__n_nationkey, DBStringType* d_nation__n_name, DBI32Type* d_nation__n_regionkey, DBStringType* d_nation__n_comment, size_t nation_size, DBI32Type* d_supplier__s_suppkey, DBI32Type* d_supplier__s_nationkey, DBStringType* d_supplier__s_name, DBStringType* d_supplier__s_address, DBStringType* d_supplier__s_phone, DBDecimalType* d_supplier__s_acctbal, DBStringType* d_supplier__s_comment, size_t supplier_size, DBI32Type* d_partsupp__ps_suppkey, DBI32Type* d_partsupp__ps_partkey, DBI32Type* d_partsupp__ps_availqty, DBDecimalType* d_partsupp__ps_supplycost, DBStringType* d_partsupp__ps_comment, size_t partsupp_size, DBI32Type* d_part__p_partkey, DBStringType* d_part__p_name, DBStringType* d_part__p_mfgr, DBStringType* d_part__p_brand, DBStringType* d_part__p_type, DBI32Type* d_part__p_size, DBStringType* d_part__p_container, DBDecimalType* d_part__p_retailprice, DBStringType* d_part__p_comment, size_t part_size, DBI32Type* d_lineitem__l_orderkey, DBI32Type* d_lineitem__l_partkey, DBI32Type* d_lineitem__l_suppkey, DBI64Type* d_lineitem__l_linenumber, DBDecimalType* d_lineitem__l_quantity, DBDecimalType* d_lineitem__l_extendedprice, DBDecimalType* d_lineitem__l_discount, DBDecimalType* d_lineitem__l_tax, DBCharType* d_lineitem__l_returnflag, DBCharType* d_lineitem__l_linestatus, DBDateType* d_lineitem__l_shipdate, DBDateType* d_lineitem__l_commitdate, DBDateType* d_lineitem__l_receiptdate, DBStringType* d_lineitem__l_shipinstruct, DBStringType* d_lineitem__l_shipmode, DBStringType* d_lineitem__comments, size_t lineitem_size, DBI32Type* d_orders__o_orderkey, DBCharType* d_orders__o_orderstatus, DBI32Type* d_orders__o_custkey, DBDecimalType* d_orders__o_totalprice, DBDateType* d_orders__o_orderdate, DBStringType* d_orders__o_orderpriority, DBStringType* d_orders__o_clerk, DBI32Type* d_orders__o_shippriority, DBStringType* d_orders__o_comment, size_t orders_size, DBI32Type* d_customer__c_custkey, DBStringType* d_customer__c_name, DBStringType* d_customer__c_address, DBI32Type* d_customer__c_nationkey, DBStringType* d_customer__c_phone, DBDecimalType* d_customer__c_acctbal, DBStringType* d_customer__c_mktsegment, DBStringType* d_customer__c_comment, size_t customer_size, DBI32Type* d_region__r_regionkey, DBStringType* d_region__r_name, DBStringType* d_region__r_comment, size_t region_size) {
//Materialize count
uint64_t* d_COUNT5b84ec4372d0;
hipMalloc(&d_COUNT5b84ec4372d0, sizeof(uint64_t));
hipMemset(d_COUNT5b84ec4372d0, 0, sizeof(uint64_t));
count_5b84ec34b570<<<std::ceil((float)customer_size/32.), 32>>>(d_COUNT5b84ec4372d0, customer_size);
uint64_t COUNT5b84ec4372d0;
hipMemcpy(&COUNT5b84ec4372d0, d_COUNT5b84ec4372d0, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5b84ec4372d0);
// Insert hash table control;
uint64_t* d_BUF_IDX_5b84ec4372d0;
hipMalloc(&d_BUF_IDX_5b84ec4372d0, sizeof(uint64_t));
hipMemset(d_BUF_IDX_5b84ec4372d0, 0, sizeof(uint64_t));
uint64_t* d_BUF_5b84ec4372d0;
hipMalloc(&d_BUF_5b84ec4372d0, sizeof(uint64_t) * COUNT5b84ec4372d0 * 1);
auto d_HT_5b84ec4372d0 = cuco::experimental::static_multimap{ (int)COUNT5b84ec4372d0*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
main_5b84ec34b570<<<std::ceil((float)customer_size/32.), 32>>>(d_BUF_5b84ec4372d0, d_BUF_IDX_5b84ec4372d0, d_HT_5b84ec4372d0.ref(cuco::insert), d_customer__c_custkey, customer_size);
hipFree(d_BUF_IDX_5b84ec4372d0);
//Create aggregation hash table
auto d_HT_5b84ec3f5bb0 = cuco::static_map{ (int)1500000*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5b84ec3cd3d0<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5b84ec4372d0, d_HT_5b84ec3f5bb0.ref(cuco::insert), d_HT_5b84ec4372d0.ref(cuco::for_each), d_customer__c_custkey, d_orders__o_custkey, orders_size);
size_t COUNT5b84ec3f5bb0 = d_HT_5b84ec3f5bb0.size();
thrust::device_vector<int64_t> keys_5b84ec3f5bb0(COUNT5b84ec3f5bb0), vals_5b84ec3f5bb0(COUNT5b84ec3f5bb0);
d_HT_5b84ec3f5bb0.retrieve_all(keys_5b84ec3f5bb0.begin(), vals_5b84ec3f5bb0.begin());
d_HT_5b84ec3f5bb0.clear();
int64_t* raw_keys5b84ec3f5bb0 = thrust::raw_pointer_cast(keys_5b84ec3f5bb0.data());
insertKeys<<<std::ceil((float)COUNT5b84ec3f5bb0/32.), 32>>>(raw_keys5b84ec3f5bb0, d_HT_5b84ec3f5bb0.ref(cuco::insert), COUNT5b84ec3f5bb0);
//Aggregate in hashtable
DBI64Type* d_aggr0__tmp_attr0;
hipMalloc(&d_aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT5b84ec3f5bb0);
hipMemset(d_aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT5b84ec3f5bb0);
DBI32Type* d_KEY_5b84ec3f5bb0customer__c_custkey;
hipMalloc(&d_KEY_5b84ec3f5bb0customer__c_custkey, sizeof(DBI32Type) * COUNT5b84ec3f5bb0);
hipMemset(d_KEY_5b84ec3f5bb0customer__c_custkey, 0, sizeof(DBI32Type) * COUNT5b84ec3f5bb0);
main_5b84ec3cd3d0<<<std::ceil((float)orders_size/32.), 32>>>(d_BUF_5b84ec4372d0, d_HT_5b84ec3f5bb0.ref(cuco::find), d_HT_5b84ec4372d0.ref(cuco::for_each), d_KEY_5b84ec3f5bb0customer__c_custkey, d_aggr0__tmp_attr0, d_customer__c_custkey, d_orders__o_custkey, d_orders__o_orderkey, orders_size);
//Create aggregation hash table
auto d_HT_5b84ec3f61d0 = cuco::static_map{ (int)1500000*2, cuco::empty_key{(int64_t)-1},cuco::empty_value{(int64_t)-1},thrust::equal_to<int64_t>{},cuco::linear_probing<1, cuco::default_hash_function<int64_t>>() };
count_5b84ec442060<<<std::ceil((float)COUNT5b84ec3f5bb0/32.), 32>>>(COUNT5b84ec3f5bb0, d_HT_5b84ec3f61d0.ref(cuco::insert), d_aggr0__tmp_attr0);
size_t COUNT5b84ec3f61d0 = d_HT_5b84ec3f61d0.size();
thrust::device_vector<int64_t> keys_5b84ec3f61d0(COUNT5b84ec3f61d0), vals_5b84ec3f61d0(COUNT5b84ec3f61d0);
d_HT_5b84ec3f61d0.retrieve_all(keys_5b84ec3f61d0.begin(), vals_5b84ec3f61d0.begin());
d_HT_5b84ec3f61d0.clear();
int64_t* raw_keys5b84ec3f61d0 = thrust::raw_pointer_cast(keys_5b84ec3f61d0.data());
insertKeys<<<std::ceil((float)COUNT5b84ec3f61d0/32.), 32>>>(raw_keys5b84ec3f61d0, d_HT_5b84ec3f61d0.ref(cuco::insert), COUNT5b84ec3f61d0);
//Aggregate in hashtable
DBI64Type* d_aggr1__tmp_attr1;
hipMalloc(&d_aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT5b84ec3f61d0);
hipMemset(d_aggr1__tmp_attr1, 0, sizeof(DBI64Type) * COUNT5b84ec3f61d0);
DBI64Type* d_KEY_5b84ec3f61d0aggr0__tmp_attr0;
hipMalloc(&d_KEY_5b84ec3f61d0aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT5b84ec3f61d0);
hipMemset(d_KEY_5b84ec3f61d0aggr0__tmp_attr0, 0, sizeof(DBI64Type) * COUNT5b84ec3f61d0);
main_5b84ec442060<<<std::ceil((float)COUNT5b84ec3f5bb0/32.), 32>>>(COUNT5b84ec3f5bb0, d_HT_5b84ec3f61d0.ref(cuco::find), d_KEY_5b84ec3f61d0aggr0__tmp_attr0, d_aggr0__tmp_attr0, d_aggr1__tmp_attr1);
//Materialize count
uint64_t* d_COUNT5b84ec409290;
hipMalloc(&d_COUNT5b84ec409290, sizeof(uint64_t));
hipMemset(d_COUNT5b84ec409290, 0, sizeof(uint64_t));
count_5b84ec443a60<<<std::ceil((float)COUNT5b84ec3f61d0/32.), 32>>>(COUNT5b84ec3f61d0, d_COUNT5b84ec409290);
uint64_t COUNT5b84ec409290;
hipMemcpy(&COUNT5b84ec409290, d_COUNT5b84ec409290, sizeof(uint64_t), hipMemcpyDeviceToHost);
hipFree(d_COUNT5b84ec409290);
//Materialize buffers
uint64_t* d_MAT_IDX5b84ec409290;
hipMalloc(&d_MAT_IDX5b84ec409290, sizeof(uint64_t));
hipMemset(d_MAT_IDX5b84ec409290, 0, sizeof(uint64_t));
auto MAT5b84ec409290aggr0__tmp_attr0 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT5b84ec409290);
DBI64Type* d_MAT5b84ec409290aggr0__tmp_attr0;
hipMalloc(&d_MAT5b84ec409290aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT5b84ec409290);
auto MAT5b84ec409290aggr1__tmp_attr1 = (DBI64Type*)malloc(sizeof(DBI64Type) * COUNT5b84ec409290);
DBI64Type* d_MAT5b84ec409290aggr1__tmp_attr1;
hipMalloc(&d_MAT5b84ec409290aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT5b84ec409290);
main_5b84ec443a60<<<std::ceil((float)COUNT5b84ec3f61d0/32.), 32>>>(COUNT5b84ec3f61d0, d_MAT5b84ec409290aggr0__tmp_attr0, d_MAT5b84ec409290aggr1__tmp_attr1, d_MAT_IDX5b84ec409290, d_KEY_5b84ec3f61d0aggr0__tmp_attr0, d_aggr1__tmp_attr1);
hipFree(d_MAT_IDX5b84ec409290);
hipMemcpy(MAT5b84ec409290aggr0__tmp_attr0, d_MAT5b84ec409290aggr0__tmp_attr0, sizeof(DBI64Type) * COUNT5b84ec409290, hipMemcpyDeviceToHost);
hipMemcpy(MAT5b84ec409290aggr1__tmp_attr1, d_MAT5b84ec409290aggr1__tmp_attr1, sizeof(DBI64Type) * COUNT5b84ec409290, hipMemcpyDeviceToHost);
for (auto i=0ull; i < COUNT5b84ec409290; i++) { std::cout << MAT5b84ec409290aggr0__tmp_attr0[i] << "\t";
std::cout << MAT5b84ec409290aggr1__tmp_attr1[i] << "\t";
std::cout << std::endl; }
}